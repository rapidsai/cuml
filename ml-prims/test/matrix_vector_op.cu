/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "matrix_vector_op.h"
#include "random/rng.h"
#include "test_utils.h"


namespace MLCommon {
namespace LinAlg {

template <typename T>
struct MatVecOpInputs {
  T tolerance;
  int rows, cols;
  bool rowMajor, bcastAlongRows, useTwoVectors;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const MatVecOpInputs<T> &dims) {
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T>
void matrixVectorOpLaunch(T *out, const T *in, const T *vec1, const T *vec2,
                          int D, int N, bool rowMajor, bool bcastAlongRows,
                          bool useTwoVectors) {
  if(useTwoVectors) {
    matrixVectorOp(out, in, vec1, vec2, D, N, rowMajor, bcastAlongRows,
                   [] __device__(T a, T b, T c) { return a + b + c; });
  } else {
    matrixVectorOp(out, in, vec1, D, N, rowMajor, bcastAlongRows,
                   [] __device__(T a, T b) { return a + b; });
  }
}

template <typename T>
class MatVecOpTest : public ::testing::TestWithParam<MatVecOpInputs<T>> {
protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MatVecOpInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int N = params.rows, D = params.cols;
    int len = N * D;
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    int vecLen = params.bcastAlongRows ? D : N;
    allocate(vec1, vecLen);
    allocate(vec2, vecLen);
    r.uniform(in, len, (T)-1.0, (T)1.0);
    r.uniform(vec1, vecLen, (T)-1.0, (T)1.0);
    r.uniform(vec2, vecLen, (T)-1.0, (T)1.0);
    if(params.useTwoVectors) {
      naiveMatVec(out_ref, in, vec1, vec2, D, N, params.rowMajor,
                  params.bcastAlongRows, (T)1.0);
    } else {
      naiveMatVec(out_ref, in, vec1, D, N, params.rowMajor,
                  params.bcastAlongRows, (T)1.0);
    }
    matrixVectorOpLaunch(out, in, vec1, vec2, D, N, params.rowMajor,
                         params.bcastAlongRows, params.useTwoVectors);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(vec1));
    CUDA_CHECK(hipFree(vec2));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(in));
  }

protected:
  MatVecOpInputs<T> params;
  T *in, *out, *out_ref, *vec1, *vec2;
};


const std::vector<MatVecOpInputs<float>> inputsf = {
  {0.00001f, 1024, 32, true, true, false, 1234ULL},
  {0.00001f, 1024, 64, true, true, false, 1234ULL},
  {0.00001f, 1024, 32, true, false, false, 1234ULL},
  {0.00001f, 1024, 64, true, false, false, 1234ULL},
  {0.00001f, 1024, 32, false, true, false, 1234ULL},
  {0.00001f, 1024, 64, false, true, false, 1234ULL},
  {0.00001f, 1024, 32, false, false, false, 1234ULL},
  {0.00001f, 1024, 64, false, false, false, 1234ULL},

  {0.00001f, 1024, 32, true, true, true, 1234ULL},
  {0.00001f, 1024, 64, true, true, true, 1234ULL},
  {0.00001f, 1024, 32, true, false, true, 1234ULL},
  {0.00001f, 1024, 64, true, false, true, 1234ULL},
  {0.00001f, 1024, 32, false, true, true, 1234ULL},
  {0.00001f, 1024, 64, false, true, true, 1234ULL},
  {0.00001f, 1024, 32, false, false, true, 1234ULL},
  {0.00001f, 1024, 64, false, false, true, 1234ULL}};
typedef MatVecOpTest<float> MatVecOpTestF;
TEST_P(MatVecOpTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestF,
                        ::testing::ValuesIn(inputsf));


const std::vector<MatVecOpInputs<double>> inputsd = {
  {0.0000001, 1024, 32, true, true, false, 1234ULL},
  {0.0000001, 1024, 64, true, true, false, 1234ULL},
  {0.0000001, 1024, 32, true, false, false, 1234ULL},
  {0.0000001, 1024, 64, true, false, false, 1234ULL},
  {0.0000001, 1024, 32, false, true, false, 1234ULL},
  {0.0000001, 1024, 64, false, true, false, 1234ULL},
  {0.0000001, 1024, 32, false, false, false, 1234ULL},
  {0.0000001, 1024, 64, false, false, false, 1234ULL},

  {0.0000001, 1024, 32, true, true, true, 1234ULL},
  {0.0000001, 1024, 64, true, true, true, 1234ULL},
  {0.0000001, 1024, 32, true, false, true, 1234ULL},
  {0.0000001, 1024, 64, true, false, true, 1234ULL},
  {0.0000001, 1024, 32, false, true, true, 1234ULL},
  {0.0000001, 1024, 64, false, true, true, 1234ULL},
  {0.0000001, 1024, 32, false, false, true, 1234ULL},
  {0.0000001, 1024, 64, false, false, true, 1234ULL}};
typedef MatVecOpTest<double> MatVecOpTestD;
TEST_P(MatVecOpTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestD,
                        ::testing::ValuesIn(inputsd));

} // end namespace LinAlg
} // end namespace MLCommon
