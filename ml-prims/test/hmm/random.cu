#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "hmm/random.h"

#include "hmm/utils.h"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cmath>

#define IDX(i,j,lda) ((i)+(j)*(lda))

namespace MLCommon {
namespace HMM {


template <typename T>
__global__ void naiveMatrixSumKernel(T* sums, T* matrix, int n_rows, int n_cols,
                                     bool colwise) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (colwise) {
                if(idx < n_rows) {
                        for( int j = 0; j < n_cols; j = j + 1 ) {
                                sums[idx] = sums[idx] + matrix[IDX(idx, j, n_rows)];
                        }
                }
        }
        else {
                if(idx < n_cols) {
                        for( int j = 0; j < n_rows; j = j + 1 ) {
                                sums[idx] += matrix[j + idx * n_rows];
                        }
                }
        }
}


template <typename T>
void naiveMatrixSum(T* sums, T* matrix, int n_rows, int n_cols, bool colwise) {
        static const int TPB = 64;
        int nblks = ceildiv(n_rows, TPB);
        naiveMatrixSumKernel<T><<<nblks,TPB>>>(sums, matrix, n_rows, n_cols, colwise);

        CUDA_CHECK(hipPeekAtLastError());
}


template <typename T>
struct error_functor
{
        __host__ __device__
        T operator()(const T& x) const {
                return std::abs(x - 1);
        }
};


template <typename T>
T compute_error(T* sums, int n_rows){
        // transfer to device
        thrust::device_ptr<T> sums_th(sums);

        // setup arguments
        error_functor<T> unary_op;
        thrust::plus<T> binary_op;
        T init = 0;

        // compute error
        T error = thrust::transform_reduce(sums_th, sums_th + n_rows,
                                           unary_op, init, binary_op);
        return error;
}

template <typename T>
struct RandomMatrixInputs {
        T tolerance;
        int n_rows, n_cols;
        unsigned long long int seed;
        int random_start, random_end;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const RandomMatrixInputs<T>& dims) {
        return os;
}

template <typename T>
class RandomMatrixTest : public ::testing::TestWithParam<RandomMatrixInputs<T> > {
protected:
void SetUp() override {
        params = ::testing::TestWithParam<RandomMatrixInputs<T> >::GetParam();
        Random::Rng<T> r(params.seed);
        tolerance = params.tolerance;
        n_rows = params.n_rows;
        n_cols = params.n_cols;
        array_size = n_rows * n_cols;
        seed = params.seed;
        paramsRandom<T> paramsRd(params.random_start, params.random_end, params.seed);

        // allocate memory
        allocate(random_matrix, array_size);
        allocate(sums_colwise, n_rows);
        allocate(sums_rowwise, n_cols);

        MLCommon::HMM::gen_array(random_matrix, array_size, &paramsRd);
        error = compute_error_type(sums_colwise, n_rows, n_cols, true);
        error += compute_error_type(sums_rowwise, n_rows, n_cols, false);

}

T compute_error_type(T* sums, int n_rows, int n_cols, bool colwise){
        thrust::device_ptr<T> sums_th(sums);
        if (colwise) {
                thrust::fill(sums_th, sums_th + n_rows, (T) 0);
        }
        else{
                thrust::fill(sums_th, sums_th + n_cols, (T) 0);
        }
        MLCommon::HMM::normalize_matrix(random_matrix, n_rows, n_cols, colwise);
        naiveMatrixSum(sums, random_matrix, n_rows, n_cols, colwise);

        if (colwise) {
                return compute_error(sums, n_rows);
        }
        else{
                return compute_error(sums, n_cols);
        }
}

void TearDown() override {
        CUDA_CHECK(hipFree(sums_colwise));
        CUDA_CHECK(hipFree(sums_rowwise));
        CUDA_CHECK(hipFree(random_matrix));
}

protected:
RandomMatrixInputs<T> params;
// random_matrix is generated with the primitive
// sums are the rowwize sums which should be equal to 1
T *random_matrix, *sums_colwise, *sums_rowwise;
int array_size;
int n_rows, n_cols;
unsigned long long seed;
T error, tolerance;
};

const std::vector<RandomMatrixInputs<float> > inputsf2 = {
        {0.000001f, 5, 3, 1234ULL, 0, 1}
};

const std::vector<RandomMatrixInputs<double> > inputsd2 = {
        {0.000001, 5, 3, 1234ULL, 0, 1}
};


typedef RandomMatrixTest<float> RandomMatrixTestF;
TEST_P(RandomMatrixTestF, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

typedef RandomMatrixTest<double> RandomMatrixTestD;
TEST_P(RandomMatrixTestD, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

INSTANTIATE_TEST_CASE_P(RandomMatrixTests, RandomMatrixTestF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(RandomMatrixTests, RandomMatrixTestD,
                        ::testing::ValuesIn(inputsd2));

} // end namespace LinAlg
} // end namespace MLCommon
