#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hmm/determinant.h"
#include "hmm/likelihood.h"
// #include "hmm/utils.h"

#include "cuda_utils.h"


using namespace MLCommon::LinAlg;
using namespace MLCommon;

namespace MLCommon {
namespace HMM {


template <typename T>
struct DeterminantInputs {
        T tolerance;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const DeterminantInputs<T>& dims) {
        return os;
}

template <typename T>
class DeterminantTest : public ::testing::TestWithParam<DeterminantInputs<T> > {
protected:
void SetUp() override {
        params = ::testing::TestWithParam<DeterminantInputs<T> >::GetParam();
        initialize(params);
        copy_to_device();
        compute_error_det(true_det);
}

void initialize(DeterminantInputs<T> params){
        tolerance = params.tolerance;
        nDim = 2;

        M_h = (T *)malloc(nDim * nDim * sizeof(T));
        allocate(M_d, nDim * nDim);
        CUDA_CHECK(hipMemset(M_d, (T)0, nDim * nDim));


        M_h[0] = (T) 9.5;
        M_h[1] = (T) 5;
        M_h[2] = (T) 5.;
        M_h[3] = (T) 4.;
        true_det = (T) 13;

        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverHandle));
        this->Det = new Determinant<T>(nDim, &cusolverHandle);
}


void copy_to_device(){
        updateDevice(M_d, M_h, nDim * nDim);
}


void compute_error_det(T true_det){
        T est_det = Det->compute(M_d);
        // printf("line number %d in file %s\n", __LINE__, __FILE__);
        error = std::abs(est_det - true_det);
}


void TearDown() override {
        free(M_h);
        CUDA_CHECK(hipFree(M_d));
        CUSOLVER_CHECK(hipsolverDnDestroy(cusolverHandle));
        Det->TearDown();
}

protected:
DeterminantInputs<T> params;

Determinant<T> *Det;

T error, tolerance;
T true_det;
int nDim;

T *M_h, *M_d;
hipsolverHandle_t cusolverHandle;
};


const std::vector<DeterminantInputs<float> > inputsf2 = {
        {0.00001f}
};

const std::vector<DeterminantInputs<double> > inputsd2 = {
        {0.00001}
};


typedef DeterminantTest<float> DeterminantTestF;
TEST_P(DeterminantTestF, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

typedef DeterminantTest<double> DeterminantTestD;
TEST_P(DeterminantTestD, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

INSTANTIATE_TEST_CASE_P(DeterminantTests, DeterminantTestF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(DeterminantTests, DeterminantTestD,
                        ::testing::ValuesIn(inputsd2));

} // end namespace HMM
} // end namespace MLCommon
