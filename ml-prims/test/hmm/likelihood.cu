#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hmm/likelihood.h"
#include "hmm/utils.h"

#include "cuda_utils.h"


using namespace MLCommon::LinAlg;
using namespace MLCommon;

namespace MLCommon {
namespace HMM {


template <typename T>
struct LikelihoodInputs {
        T tolerance;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const LikelihoodInputs<T>& dims) {
        return os;
}

template <typename T>
class LikelihoodTest : public ::testing::TestWithParam<LikelihoodInputs<T> > {
protected:
void SetUp() override {
        params = ::testing::TestWithParam<LikelihoodInputs<T> >::GetParam();
        tolerance = params.tolerance;

        initialize_parameters();
        allocate_memory();
        initialize();
        copy_to_device();

        error = compute_error_lhd();
}

T _compute_error_dist(int distId){
        T est_val, true_val;
        // if (distId == 0) {
        true_val  = _naive_lhd();
        est_val = set_gmm_lhd(data_d, mus_d, sigmas_d, rhos_d, nCl,isLog, nDim, nObs, &handle);
        // est_val = _sample_gaussian_lhd(data_d, mus_d, sigmas_d, nDim, handle);
        // }
        return std::abs(true_val - est_val);
}

T compute_error_lhd(){
        T diff=0;
        for (int distId = 0; distId < nDists; distId++) {
                diff += _compute_error_dist(distId);
        }
        diff /= nDists;
        return diff;
}

T _naive_lhd(){
        return (T) 0.0;
}


void allocate_memory(){
        allocate(data_d, nDim * nObs);
        CUDA_CHECK(hipMemset(data_d, (T)0, nDim * nObs ));
        data_h = (T *)malloc(nDim * nObs * sizeof(T));
        print_matrix(data_d, nDim, nObs, "data_d");


        allocate(mus_d, nDim * nCl);
        CUDA_CHECK(hipMemset(mus_d, (T)0,nDim * nCl ));
        mus_h = (T *)malloc(nDim * nCl * sizeof(T));

        allocate(sigmas_d, nDim *nDim * nCl);
        CUDA_CHECK(hipMemset(sigmas_d, (T) 0, nDim *nDim * nCl));
        sigmas_h = (T *)malloc(nDim * nDim * nCl * sizeof(T));

        allocate(rhos_d, nCl);
        CUDA_CHECK(hipMemset(rhos_d, (T) 0, nCl));
        rhos_h = (T *)malloc(nCl * sizeof(T));

        CUBLAS_CHECK(hipblasCreate(&handle));
}

void initialize_parameters(){
        nDim = 2;
        nCl = 2;
        nObs = 2;
        nDists = 1;
        isLog = true;
}

void initialize(){

        data_h[0] = 0.0;
        data_h[1] = -1.0;
        data_h[2] = 1.0;
        data_h[3] = 5.0;

        mus_h[0] = -1.0;
        mus_h[1] = 2.0;
        mus_h[2] = 1.0;
        mus_h[3] = 1.0;

        rhos_h[0] = 0.3;
        rhos_h[1] = 0.7;

        sigmas_h[0] = 1;
        sigmas_h[1] = 0.0;
        sigmas_h[2] = 0.0;
        sigmas_h[3] = 1.0;
        sigmas_h[4] = 1.0;
        sigmas_h[5] = 0.5;
        sigmas_h[6] = 0.7;
        sigmas_h[7] = -2.0;
}

void copy_to_device(){
        updateDevice(data_d, data_h, nDim * nObs);
        updateDevice(mus_d, mus_h, nDim * nCl);
        updateDevice(sigmas_d, sigmas_h, nDim *nDim * nCl);
        updateDevice(rhos_d, rhos_h, nCl);
}



void TearDown() override {
        free(data_h);
        free(mus_h);
        free(sigmas_h);
        free(rhos_h);

        CUDA_CHECK(hipFree(data_d));
        CUDA_CHECK(hipFree(mus_d));
        CUDA_CHECK(hipFree(sigmas_d));
        CUDA_CHECK(hipFree(rhos_d));
}

protected:
LikelihoodInputs<T> params;
T error, tolerance;
int nDim, nCl, nObs, nDists;
bool isLog;

T *data_h, *mus_h, *sigmas_h, *rhos_h;
T *data_d, *mus_d, *sigmas_d, *rhos_d;
hipblasHandle_t handle;
};

const std::vector<LikelihoodInputs<float> > inputsf2 = {
        {0.00001f}
};

const std::vector<LikelihoodInputs<double> > inputsd2 = {
        {0.00001}
};


typedef LikelihoodTest<float> LikelihoodTestF;
TEST_P(LikelihoodTestF, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

typedef LikelihoodTest<double> LikelihoodTestD;
TEST_P(LikelihoodTestD, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

INSTANTIATE_TEST_CASE_P(LikelihoodTests, LikelihoodTestF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(LikelihoodTests, LikelihoodTestD,
                        ::testing::ValuesIn(inputsd2));

} // end namespace HMM
} // end namespace MLCommon
