/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "linalg/unary_op.h"
#include "random/rng.h"
#include "test_utils.h"
#include "unary_op.h"


namespace MLCommon {
namespace LinAlg {

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T>
void unaryOpLaunch(T *out, const T *in, T scalar, int len) {
  unaryOp(out, in, len,
          [scalar] __device__(T in) { return in * scalar; });
}

template <typename T>
class UnaryOpTest : public ::testing::TestWithParam<UnaryOpInputs<T>> {
protected:
  void SetUp() override {
    params = ::testing::TestWithParam<UnaryOpInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.len;
    T scalar = params.scalar;
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in, len, T(-1.0), T(1.0));
    naiveScale(out_ref, in, scalar, len);
    unaryOpLaunch(out, in, scalar, len);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

protected:
  UnaryOpInputs<T> params;
  T *in, *out_ref, *out;
};

const std::vector<UnaryOpInputs<float>> inputsf = {
  {0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef UnaryOpTest<float> UnaryOpTestF;
TEST_P(UnaryOpTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(UnaryOpTests, UnaryOpTestF,
                        ::testing::ValuesIn(inputsf));

const std::vector<UnaryOpInputs<double>> inputsd = {
  {0.00000001, 1024 * 1024, 2.0, 1234ULL}};
typedef UnaryOpTest<double> UnaryOpTestD;
TEST_P(UnaryOpTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(UnaryOpTests, UnaryOpTestD,
                        ::testing::ValuesIn(inputsd));

} // end namespace LinAlg
} // end namespace MLCommon
