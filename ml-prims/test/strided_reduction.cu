/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "linalg/strided_reduction.h"
#include "linalg/unary_op.h"
#include "random/rng.h"
#include "test_utils.h"

#include <thrust/device_vector.h>
#include <hipblas.h>

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct stridedReductionInputs {
    T tolerance;
    int rows, cols;
    unsigned long long int seed;
};

template <typename T>
void stridedReductionLaunch(T *dots, const T *data, int cols, int rows) {
  stridedReduction(dots, data, cols, rows, (T)0, false, 0,
                   [] __device__(T in) { return in * in; });
}


template <typename T, typename GEMV_t>
void unaryAndGemv(T *dots, const T *data, int cols, int rows, GEMV_t gemv){
    //computes a MLCommon unary op on data (squares it), then computes Ax
    //(A input matrix and x column vector) to sum columns
    thrust::device_vector<T> sq(cols*rows);
    unaryOp(thrust::raw_pointer_cast(sq.data()), data, cols*rows,
            [] __device__(T v) { return v*v; });

    hipblasHandle_t handle;
    ASSERT_TRUE(hipblasCreate(&handle) == HIPBLAS_STATUS_SUCCESS);

    thrust::device_vector<T> ones(rows, 1); //column vector [1...1]
    T alpha = 1, beta = 0;
    ASSERT_TRUE(gemv(handle, HIPBLAS_OP_N, cols, rows,
                &alpha, thrust::raw_pointer_cast(sq.data()), cols,
                thrust::raw_pointer_cast(ones.data()), 1, &beta, 
                dots, 1) == HIPBLAS_STATUS_SUCCESS);
}

void unaryAndGemv(float *dots, const float *data, int cols, int rows){
    unaryAndGemv(dots, data, cols, rows, hipblasSgemv);
}

void unaryAndGemv(double *dots, const double *data, int cols, int rows){
    unaryAndGemv(dots, data, cols, rows, hipblasDgemv);
}


template <typename T>
class stridedReductionTest : public ::testing::TestWithParam<stridedReductionInputs<T>> {
protected:
  void SetUp() override {
    params = ::testing::TestWithParam<stridedReductionInputs<T>>::GetParam();
    Random::Rng<T> r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows*cols;

    allocate(data, len);
    allocate(dots_exp, cols); //expected dot products (from test)
    allocate(dots_act, cols); //actual dot products (from prim)
    r.uniform(data, len, -1.f, 1.f); //initialize matrix to random

    unaryAndGemv(dots_exp, data, cols, rows);
    stridedReductionLaunch(dots_act, data, cols, rows);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(dots_exp));
    CUDA_CHECK(hipFree(dots_act));
  }

protected:
  stridedReductionInputs<T> params;
  T *data, *dots_exp, *dots_act;
};


const std::vector<stridedReductionInputs<float>> inputsf = {
  {0.00001f, 1024,  32, 1234ULL},
  {0.00001f, 1024,  64, 1234ULL},
  {0.00001f, 1024, 128, 1234ULL},
  {0.00001f, 1024, 256, 1234ULL},
  {0.00001f, 1024,  32, 1234ULL},
  {0.00001f, 1024,  64, 1234ULL},
  {0.00001f, 1024, 128, 1234ULL},
  {0.00001f, 1024, 256, 1234ULL}
};

const std::vector<stridedReductionInputs<double>> inputsd = {
  {0.000000001, 1024,  32, 1234ULL},
  {0.000000001, 1024,  64, 1234ULL},
  {0.000000001, 1024, 128, 1234ULL},
  {0.000000001, 1024, 256, 1234ULL},
  {0.000000001, 1024,  32, 1234ULL},
  {0.000000001, 1024,  64, 1234ULL},
  {0.000000001, 1024, 128, 1234ULL},
  {0.000000001, 1024, 256, 1234ULL}
};

typedef stridedReductionTest<float> stridedReductionTestF;
TEST_P(stridedReductionTestF, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef stridedReductionTest<double> stridedReductionTestD;
TEST_P(stridedReductionTestD, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(stridedReductionTests, stridedReductionTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(stridedReductionTests, stridedReductionTestD, ::testing::ValuesIn(inputsd));

} // end namespace LinAlg
} // end namespace MLCommon
