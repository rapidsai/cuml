/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "add.h"
#include "linalg/add.h"
#include "random/rng.h"
#include "test_utils.h"


namespace MLCommon {
namespace LinAlg {


template <typename T>
class AddTest : public ::testing::TestWithParam<AddInputs<T>> {
protected:
  void SetUp() override {
    params = ::testing::TestWithParam<AddInputs<T>>::GetParam();
    Random::Rng<T> r(params.seed);
    int len = params.len;
    allocate(in1, len);
    allocate(in2, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in1, len, T(-1.0), T(1.0));
    r.uniform(in2, len, T(-1.0), T(1.0));
    naiveAddElem(out_ref, in1, in2, len);
    add(out, in1, in2, len);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

protected:
  AddInputs<T> params;
  T *in1, *in2, *out_ref, *out;
};


const std::vector<AddInputs<float>> inputsf2 = {
  {0.000001f, 1024 * 1024, 1234ULL}};
typedef AddTest<float> AddTestF;
TEST_P(AddTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(AddTests, AddTestF, ::testing::ValuesIn(inputsf2));


const std::vector<AddInputs<double>> inputsd2 = {
  {0.00000001, 1024 * 1024, 1234ULL}};
typedef AddTest<double> AddTestD;
TEST_P(AddTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(AddTests, AddTestD, ::testing::ValuesIn(inputsd2));

} // end namespace LinAlg
} // end namespace MLCommon
