/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "csr.h"
#include <gtest/gtest.h>
#include "sparse/csr.h"
#include "random/rng.h"
#include "test_utils.h"

#include <iostream>

namespace MLCommon {
namespace Sparse {

template <typename T>
class CSRTest : public ::testing::TestWithParam<CSRInputs<T>> {
protected:
  void SetUp() override {}

  void TearDown() override {}

protected:
  CSRInputs<T> params;
};

const std::vector<CSRInputs<float>> inputsf = {
  {5, 10, 5, 1234ULL}};

typedef CSRTest<float> CSRRowNormalizeL1;
TEST_P(CSRRowNormalizeL1, Result) {

    int *ex_scan;
    float *in_vals, *result, *verify;

    int ex_scan_h[4] = {0, 4, 8, 9 };
    float in_vals_h[10] = { 1.0, 1.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0 };

    float verify_h[10] =  { 0.5, 0.5, 0.0, 0.0, 0.2, 0.2, 0.0, 0.0, 0.2, 0.0 };

    allocate(in_vals, 10);
    allocate(verify, 10);
    allocate(ex_scan, 4);
    allocate(result, 10, true);

    updateDevice(ex_scan, *&ex_scan_h, 4);
    updateDevice(in_vals, *&in_vals_h, 10);
    updateDevice(verify, *&verify_h, 10);

    dim3 grid(ceildiv(10, 32), 1, 1);
    dim3 blk(32, 1, 1);

    csr_row_normalize_l1<32, float><<<grid, blk>>>(ex_scan, in_vals, 10, 10, result);
    hipDeviceSynchronize();

    ASSERT_TRUE(devArrMatch<float>(verify, result, 10, Compare<float>()));

    CUDA_CHECK(hipFree(ex_scan));
    CUDA_CHECK(hipFree(in_vals));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
}


INSTANTIATE_TEST_CASE_P(CSRTests, CSRRowNormalizeL1,
                        ::testing::ValuesIn(inputsf));

}}

