#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "cuda_utils.h"
#include "distance/distance.h"
#include "random/rng.h"
#include "test_utils.h"


namespace MLCommon {
namespace Distance {

template <typename Type>
__global__ void naiveDistanceKernel(Type *dist, const Type *x, const Type *y,
                                    int m, int n, int k, DistanceType type) {
  int midx = threadIdx.x + blockIdx.x * blockDim.x;
  int nidx = threadIdx.y + blockIdx.y * blockDim.y;
  if (midx >= m || nidx >= n)
    return;
  Type acc = Type(0);
  for (int i = 0; i < k; ++i) {
    auto diff = x[i + midx * k] - y[i + nidx * k];
    acc += diff * diff;
  }
  if (type == EucExpandedL2Sqrt || type == EucUnexpandedL2Sqrt)
    acc = mySqrt(acc);
  dist[midx * n + nidx] = acc;
}

template <typename Type>
__global__ void naiveL1DistanceKernel(Type *dist, const Type *x, const Type *y,
                                      int m, int n, int k) {
  int midx = threadIdx.x + blockIdx.x * blockDim.x;
  int nidx = threadIdx.y + blockIdx.y * blockDim.y;
  if (midx >= m || nidx >= n) {
    return;
  }

  Type acc = Type(0);
  for (int i = 0; i < k; ++i) {
    auto a = x[i + midx * k];
    auto b = y[i + nidx * k];
    auto diff = (a > b) ? (a - b) : (b - a);
    acc += diff;
  }

  dist[midx * n + nidx] = acc;
}

template <typename Type>
__global__ void naiveCosineDistanceKernel(Type *dist, const Type *x,
                                          const Type *y, int m, int n, int k) {
  int midx = threadIdx.x + blockIdx.x * blockDim.x;
  int nidx = threadIdx.y + blockIdx.y * blockDim.y;
  if (midx >= m || nidx >= n) {
    return;
  }

  Type acc_a = Type(0);
  Type acc_b = Type(0);
  Type acc_ab = Type(0);

  for (int i = 0; i < k; ++i) {
    auto a = x[i + midx * k];
    auto b = y[i + nidx * k];

    acc_a += a * a;
    acc_b += b * b;
    acc_ab += a * b;
  }

  dist[midx * n + nidx] = acc_ab / (mySqrt(acc_a) * mySqrt(acc_b));
}

template <typename Type>
void naiveDistance(Type *dist, const Type *x, const Type *y, int m, int n,
                   int k, DistanceType type) {
  static const dim3 TPB(16, 32, 1);
  dim3 nblks(ceildiv(m, (int)TPB.x), ceildiv(n, (int)TPB.y), 1);

  switch (type) {
    case EucUnexpandedL1:
      naiveL1DistanceKernel<Type><<<nblks, TPB>>>(dist, x, y, m, n, k);
      break;
    case EucUnexpandedL2Sqrt:
    case EucUnexpandedL2:
    case EucExpandedL2Sqrt:
    case EucExpandedL2:
      naiveDistanceKernel<Type><<<nblks, TPB>>>(dist, x, y, m, n, k, type);
      break;
    case EucExpandedCosine:
      naiveCosineDistanceKernel<Type><<<nblks, TPB>>>(dist, x, y, m, n, k);
      break;
    default:
      FAIL() << "should be here\n";
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct DistanceInputs {
  T tolerance;
  int m, n, k;
  DistanceType type;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const DistanceInputs<T> &dims) {
  return os;
}

template <typename T, typename OutputTile_t>
void distanceLauncher(T* x, T* y, T* dist, T* dist2, int m, int n, int k,
                      DistanceInputs<T>& params, T threshold, char* workspace,
                      size_t worksize) {
    auto fin_op = [dist2, threshold] __device__(T d_val, int g_d_idx) {
      dist2[g_d_idx] = (d_val < threshold) ? 0.f : d_val;
      return d_val;
    };
    distance<T, T, T, OutputTile_t>(
      x, y, dist, m, n, k, params.type, workspace, worksize, fin_op);
}

template <typename T>
class DistanceTest : public ::testing::TestWithParam<DistanceInputs<T>> {
public:
  void SetUp() override {
    params = ::testing::TestWithParam<DistanceInputs<T>>::GetParam();
    Random::Rng<T> r(params.seed);
    int m = params.m;
    int n = params.n;
    int k = params.k;
    allocate(x, m * k);
    allocate(y, n * k);
    allocate(dist_ref, m * n);
    allocate(dist, m * n);
    allocate(dist2, m * n);
    r.uniform(x, m * k, T(-1.0), T(1.0));
    r.uniform(y, n * k, T(-1.0), T(1.0));
    naiveDistance(dist_ref, x, y, m, n, k, params.type);
    char *workspace = nullptr;
    size_t worksize = 0;

    typedef cutlass::Shape<8, 128, 128> OutputTile_t;

    if (params.type <= EucExpandedCosine) {
      distance<T, T, T, OutputTile_t>(
        x, y, dist, m, n, k, params.type, nullptr, worksize);
      if (worksize != 0) {
        allocate(workspace, worksize);
      }
    }

    T threshold = -10000.f;
    distanceLauncher<T,OutputTile_t>(x, y, dist, dist2, m, n, k, params,
                                     threshold, workspace, worksize);
    if (params.type <= EucExpandedCosine) {
      CUDA_CHECK(hipFree(workspace));
    }
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(x));
    CUDA_CHECK(hipFree(y));
    CUDA_CHECK(hipFree(dist_ref));
    CUDA_CHECK(hipFree(dist));
    CUDA_CHECK(hipFree(dist2));
  }

protected:
  DistanceInputs<T> params;
  T *x, *y, *dist_ref, *dist, *dist2;
};

const std::vector<DistanceInputs<float>> inputsf = {
  {0.001f, 1024, 1024, 32, EucExpandedL2,
   1234ULL}, // accumulate issue due to x^2 + y^2 -2xy
  {0.001f, 1024, 32, 1024, EucExpandedL2, 1234ULL},
  {0.001f, 32, 1024, 1024, EucExpandedL2, 1234ULL},
  {0.003f, 1024, 1024, 1024, EucExpandedL2, 1234ULL},

  {0.03f, 1024, 1024, 32, EucExpandedL2Sqrt, 1234ULL},
  {0.03f, 1024, 32, 1024, EucExpandedL2Sqrt, 1234ULL},
  {0.03f, 32, 1024, 1024, EucExpandedL2Sqrt, 1234ULL},
  {0.03f, 1024, 1024, 1024, EucExpandedL2Sqrt, 1234ULL},

  {0.001f, 1024, 1024, 32, EucUnexpandedL2, 1234ULL},
  {0.001f, 1024, 32, 1024, EucUnexpandedL2, 1234ULL},
  {0.001f, 32, 1024, 1024, EucUnexpandedL2, 1234ULL},
  {0.001f, 1024, 1024, 1024, EucUnexpandedL2, 1234ULL},

  {0.001f, 1024, 1024, 32, EucUnexpandedL2Sqrt, 1234ULL},
  {0.001f, 1024, 32, 1024, EucUnexpandedL2Sqrt, 1234ULL},
  {0.001f, 32, 1024, 1024, EucUnexpandedL2Sqrt, 1234ULL},
  {0.001f, 1024, 1024, 1024, EucUnexpandedL2Sqrt, 1234ULL},

  {0.001f, 1024, 1024, 32, EucExpandedCosine, 1234ULL},
  {0.001f, 1024, 32, 1024, EucExpandedCosine, 1234ULL},
  {0.001f, 32, 1024, 1024, EucExpandedCosine, 1234ULL},
  {0.001f, 1024, 1024, 1024, EucExpandedCosine, 1234ULL},

  {0.001f, 1024, 1024, 32, EucUnexpandedL1, 1234ULL},
  {0.001f, 1024, 32, 1024, EucUnexpandedL1, 1234ULL},
  {0.001f, 32, 1024, 1024, EucUnexpandedL1, 1234ULL},
  {0.001f, 1024, 1024, 1024, EucUnexpandedL1, 1234ULL},
};
typedef DistanceTest<float> DistanceTestF;
TEST_P(DistanceTestF, Result) {
  ASSERT_TRUE(devArrMatch(dist_ref, dist, params.m, params.n,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(DistanceTests, DistanceTestF,
                        ::testing::ValuesIn(inputsf));


const std::vector<DistanceInputs<double>> inputsd = {
  {0.001, 1024, 1024, 32, EucExpandedL2,
   1234ULL}, // accumulate issue due to x^2 + y^2 -2xy
  {0.001, 1024, 32, 1024, EucExpandedL2, 1234ULL},
  {0.001, 32, 1024, 1024, EucExpandedL2, 1234ULL},
  {0.003, 1024, 1024, 1024, EucExpandedL2, 1234ULL},

  {0.03, 1024, 1024, 32, EucExpandedL2Sqrt, 1234ULL},
  {0.03, 1024, 32, 1024, EucExpandedL2Sqrt, 1234ULL},
  {0.03, 32, 1024, 1024, EucExpandedL2Sqrt, 1234ULL},
  {0.03, 1024, 1024, 1024, EucExpandedL2Sqrt, 1234ULL},

  {0.001, 1024, 1024, 32, EucUnexpandedL2, 1234ULL},
  {0.001, 1024, 32, 1024, EucUnexpandedL2, 1234ULL},
  {0.001, 32, 1024, 1024, EucUnexpandedL2, 1234ULL},
  {0.001, 1024, 1024, 1024, EucUnexpandedL2, 1234ULL},

  {0.001, 1024, 1024, 32, EucUnexpandedL2Sqrt, 1234ULL},
  {0.001, 1024, 32, 1024, EucUnexpandedL2Sqrt, 1234ULL},
  {0.001, 32, 1024, 1024, EucUnexpandedL2Sqrt, 1234ULL},
  {0.001, 1024, 1024, 1024, EucUnexpandedL2Sqrt, 1234ULL},

  {0.001, 1024, 1024, 32, EucExpandedCosine, 1234ULL},
  {0.001, 1024, 32, 1024, EucExpandedCosine, 1234ULL},
  {0.001, 32, 1024, 1024, EucExpandedCosine, 1234ULL},
  {0.001, 1024, 1024, 1024, EucExpandedCosine, 1234ULL},

  {0.001, 1024, 1024, 32, EucUnexpandedL1, 1234ULL},
  {0.001, 1024, 32, 1024, EucUnexpandedL1, 1234ULL},
  {0.001, 32, 1024, 1024, EucUnexpandedL1, 1234ULL},
  {0.001, 1024, 1024, 1024, EucUnexpandedL1, 1234ULL},
};
typedef DistanceTest<double> DistanceTestD;
TEST_P(DistanceTestD, Result) {
  ASSERT_TRUE(devArrMatch(dist_ref, dist, params.m, params.n,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(DistanceTests, DistanceTestD,
                        ::testing::ValuesIn(inputsd));

} // end namespace Distance
} // end namespace MLCommon
