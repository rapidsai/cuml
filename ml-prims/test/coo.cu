/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "coo.h"
#include <gtest/gtest.h>
#include "sparse/coo.h"
#include "random/rng.h"
#include "test_utils.h"

#include <iostream>

namespace MLCommon {
namespace Sparse {

template <typename T>
class COOTest : public ::testing::TestWithParam<COOInputs<T>> {
protected:
  void SetUp() override {}

  void TearDown() override {}

protected:
  COOInputs<T> params;
};

const std::vector<COOInputs<float>> inputsf = {
  {5, 10, 5, 1234ULL}};

typedef COOTest<float> COOSort;
TEST_P(COOSort, Result) {

    int *in_rows, *in_cols, *verify;
    float *in_vals;

    params = ::testing::TestWithParam<COOInputs<float>>::GetParam();
    Random::Rng r(params.seed);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in_vals, params.nnz);
    r.uniform(in_vals, params.nnz, float(-1.0), float(1.0), stream);

    int *in_rows_h = (int*)malloc(params.nnz * sizeof(int));
    int *in_cols_h = (int*)malloc(params.nnz * sizeof(int));
    int *verify_h = (int*)malloc(params.nnz * sizeof(int));

    for(int i = 0; i < params.nnz; i++) {
        in_rows_h[i] = params.nnz-i-1;
        verify_h[i] = i;
        in_cols_h[i] = i;
    }

    allocate(in_rows, params.nnz);
    allocate(in_cols, params.nnz);
    allocate(verify, params.nnz);

    updateDevice(in_rows, in_rows_h, params.nnz);

    updateDevice(in_cols, in_cols_h, params.nnz);
    updateDevice(verify, verify_h, params.nnz);

    coo_sort(params.m, params.n, params.nnz, in_rows, in_cols, in_vals);

    ASSERT_TRUE(devArrMatch<int>(verify, in_rows, params.nnz, Compare<int>()));

    CUDA_CHECK(hipFree(in_rows));
    CUDA_CHECK(hipFree(in_cols));
    CUDA_CHECK(hipFree(in_vals));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipStreamDestroy(stream));
}

typedef COOTest<float> COORemoveZeros;
TEST_P(COORemoveZeros, Result) {

    int *in_rows, *in_cols;
    float *in_vals;

    params = ::testing::TestWithParam<COOInputs<float>>::GetParam();
    Random::Rng r(params.seed);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in_vals, params.nnz);
    r.uniform(in_vals, params.nnz, float(-1.0), float(1.0), stream);

    float *vals_h = (float*)malloc(params.nnz * sizeof(float));
    updateHost(vals_h, in_vals, params.nnz);

    vals_h[0] = 0;
    vals_h[2] = 0;
    vals_h[3] = 0;

    updateDevice(in_vals, vals_h, params.nnz);

    int out_rows_ref_h[2]  = { 3, 0 };
    int out_cols_ref_h[2] =  { 1, 4 };

    float *out_vals_ref_h = (float*)malloc(2*sizeof(float));
    out_vals_ref_h[0] = vals_h[1];
    out_vals_ref_h[1] = vals_h[4];

    int *out_rows, *out_cols, *out_rows_ref, *out_cols_ref;
    float *out_vals, *out_vals_ref;
    allocate(out_rows, 2);
    allocate(out_cols, 2);
    allocate(out_vals, 2);
    allocate(out_rows_ref, 2);
    allocate(out_vals_ref, 2);
    allocate(out_cols_ref, 2);

    updateDevice(out_rows_ref, *&out_rows_ref_h, 2);
    updateDevice(out_cols_ref, *&out_cols_ref_h, 2);
    updateDevice(out_vals_ref, out_vals_ref_h, 2);

    int *cnnz, cnnz_h[5] = { 0, 1, 0, 0, 1 };
    allocate(cnnz, 5);

    updateDevice(cnnz, *&cnnz_h, 5);

    int *in_rows_h = (int*)malloc(params.nnz * sizeof(int));
    int *in_cols_h = (int*)malloc(params.nnz * sizeof(int));

    for(int i = 0; i < params.nnz; i++) {
        in_rows_h[i] = params.nnz-i-1;
        in_cols_h[i] = i;
    }

    allocate(in_rows, params.nnz);
    allocate(in_cols, params.nnz);

    updateDevice(in_rows, in_rows_h, params.nnz);
    updateDevice(in_cols, in_cols_h, params.nnz);

    coo_remove_zeros<16, float>(params.nnz, in_rows, in_cols, in_vals, out_rows, out_cols, out_vals, cnnz, 5);

    ASSERT_TRUE(devArrMatch<int>(out_rows_ref, out_rows, 2, Compare<int>()));
    ASSERT_TRUE(devArrMatch<int>(out_cols_ref, out_cols, 2, Compare<int>()));
    ASSERT_TRUE(devArrMatch<float>(out_vals_ref, out_vals, 2, Compare<float>()));

    CUDA_CHECK(hipFree(in_rows));
    CUDA_CHECK(hipFree(in_cols));
    CUDA_CHECK(hipFree(in_vals));
    CUDA_CHECK(hipFree(out_rows));
    CUDA_CHECK(hipFree(out_cols));
    CUDA_CHECK(hipFree(out_vals));
    CUDA_CHECK(hipFree(out_rows_ref));
    CUDA_CHECK(hipFree(out_cols_ref));
    CUDA_CHECK(hipFree(out_vals_ref));
    CUDA_CHECK(hipStreamDestroy(stream));
    free(out_vals_ref_h);
    free(in_rows_h);
    free(in_cols_h);
    free(vals_h);
}


typedef COOTest<float> COORowCount;
TEST_P(COORowCount, Result) {

    int *in_rows,*verify, *results;

    int in_rows_h[5] = { 0, 0, 1, 2, 2 };
    int verify_h[5] = {2, 1, 2, 0, 0};

    allocate(in_rows, 5);
    allocate(verify, 5, true);
    allocate(results, 5, true);

    updateDevice(in_rows, *&in_rows_h, 5);
    updateDevice(verify, *&verify_h, 5);

    dim3 grid(ceildiv(5, 32), 1, 1);
    dim3 blk(32, 1, 1);
    coo_row_count<32, float><<<grid, blk>>>(in_rows, 5, results, 5);
    hipDeviceSynchronize();

    ASSERT_TRUE(devArrMatch<int>(verify, results, 5, Compare<int>()));

    CUDA_CHECK(hipFree(in_rows));
    CUDA_CHECK(hipFree(verify));
}

typedef COOTest<float> COORowCountNonzero;
TEST_P(COORowCountNonzero, Result) {

    int *in_rows,*verify, *results;
    float *in_vals;

    int in_rows_h[5] = { 0, 0, 1, 2, 2 };
    float in_vals_h[5] = { 0.0, 5.0, 0.0, 1.0, 1.0 };
    int verify_h[5] = {1, 0, 2, 0, 0};

    allocate(in_rows, 5);
    allocate(verify, 5, true);
    allocate(results, 5, true);
    allocate(in_vals, 5, true);

    updateDevice(in_rows, *&in_rows_h, 5);
    updateDevice(verify, *&verify_h, 5);
    updateDevice(in_vals, *&in_vals_h, 5);

    dim3 grid(ceildiv(5, 32), 1, 1);
    dim3 blk(32, 1, 1);
    coo_row_count_nz<32, float><<<grid, blk>>>(in_rows, in_vals, 5, results, 5);
    hipDeviceSynchronize();

    ASSERT_TRUE(devArrMatch<int>(verify, results, 5, Compare<int>()));

    CUDA_CHECK(hipFree(in_rows));
    CUDA_CHECK(hipFree(verify));
}


INSTANTIATE_TEST_CASE_P(COOTests, COOSort,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(COOTests, COORemoveZeros,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(COOTests, COORowCount,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(COOTests, COORowCountNonzero,
                        ::testing::ValuesIn(inputsf));
}
}
