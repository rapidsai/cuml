#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <cuml/cuml.hpp>
#include <cuml/tsa/batched_arima.hpp>
#include <cuml/tsa/batched_kalman.hpp>

#include <common/cudart_utils.h>
#include <common/cumlHandle.hpp>
#include <common/device_buffer.hpp>
#include <common/nvtx.hpp>
#include <cuda_utils.cuh>
#include <linalg/batched/matrix.cuh>
#include <linalg/matrix_vector_op.cuh>
#include <metrics/batched/information_criterion.cuh>
#include <timeSeries/arima_helpers.cuh>

namespace ML {

void batched_diff(cumlHandle& handle, double* d_y_diff, const double* d_y,
                  int batch_size, int n_obs, const ARIMAOrder& order) {
  const auto stream = handle.getStream();
  MLCommon::TimeSeries::prepare_data(d_y_diff, d_y, batch_size, n_obs, order.d,
                                     order.D, order.s, stream);
}

void predict(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
             int start, int end, const ARIMAOrder& order,
             const ARIMAParams<double>& params, double* d_vs, double* d_y_p) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  const auto stream = handle.getStream();

  // Prepare data
  int diff_obs = order.lost_in_diff();
  int ld_yprep = n_obs - diff_obs;
  double* d_y_prep = (double*)allocator->allocate(
    ld_yprep * batch_size * sizeof(double), stream);
  MLCommon::TimeSeries::prepare_data(d_y_prep, d_y, batch_size, n_obs, order.d,
                                     order.D, order.s, stream);

  // Create temporary array for the forecasts
  int num_steps = std::max(end - n_obs, 0);
  double* d_y_fc = nullptr;
  if (num_steps) {
    d_y_fc = (double*)allocator->allocate(
      num_steps * batch_size * sizeof(double), stream);
  }

  // Compute the residual and forecast - provide already prepared data and
  // extracted parameters
  ARIMAOrder order_after_prep = {order.p, 0,       order.q, order.P,
                                 0,       order.Q, order.s, order.k};
  std::vector<double> loglike = std::vector<double>(batch_size);
  /// TODO: use device loglike to avoid useless copy ; part of #2233
  batched_loglike(handle, d_y_prep, batch_size, n_obs - diff_obs,
                  order_after_prep, params, loglike.data(), d_vs, false, true,
                  MLE, 0, num_steps, d_y_fc);

  auto counting = thrust::make_counting_iterator(0);
  int predict_ld = end - start;

  //
  // In-sample prediction
  //

  int p_start = std::max(start, diff_obs);
  int p_end = std::min(n_obs, end);

  // The prediction loop starts by filling undefined predictions with NaN,
  // then computes the predictions from the observations and residuals
  if (start < n_obs) {
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_y_p[0] = 0.0;
                       for (int i = 0; i < diff_obs - start; i++) {
                         d_y_p[bid * predict_ld + i] = nan("");
                       }
                       for (int i = p_start; i < p_end; i++) {
                         d_y_p[bid * predict_ld + i - start] =
                           d_y[bid * n_obs + i] -
                           d_vs[bid * ld_yprep + i - diff_obs];
                       }
                     });
  }

  //
  // Finalize out-of-sample forecast and copy in-sample predictions
  //

  if (num_steps) {
    // Add trend and/or undiff
    MLCommon::TimeSeries::finalize_forecast(d_y_fc, d_y, num_steps, batch_size,
                                            n_obs, n_obs, order.d, order.D,
                                            order.s, stream);

    // Copy forecast in d_y_p
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       for (int i = 0; i < num_steps; i++) {
                         d_y_p[bid * predict_ld + n_obs - start + i] =
                           d_y_fc[num_steps * bid + i];
                       }
                     });

    allocator->deallocate(d_y_fc, num_steps * batch_size * sizeof(double),
                          stream);
  }

  allocator->deallocate(d_y_prep, ld_yprep * batch_size * sizeof(double),
                        stream);
  ML::POP_RANGE();
}

/**
 * Kernel to compute the sum-of-squares log-likelihood estimation
 *
 * @param[in]  d_y        Series to fit
 * @param[in]  d_mu       mu parameters
 * @param[in]  d_ar       AR parameters
 * @param[in]  d_ma       MA parameters
 * @param[in]  d_sar      Seasonal AR parameters
 * @param[in]  d_sma      Seasonal MA parameters
 * @param[out] d_loglike  Evaluated log-likelihood
 * @param[in]  n_obs      Number of observations in a time series
 * @param[in]  n_phi      Number of phi coefficients (combined AR-SAR)
 * @param[in]  n_theta    Number of theta coefficients (combined MA-SMA)
 * @param[in]  p          Number of AR parameters
 * @param[in]  q          Number of MA parameters
 * @param[in]  P          Number of seasonal AR parameters
 * @param[in]  Q          Number of seasonal MA parameters
 * @param[in]  s          Seasonal period or 0
 * @param[in]  k          Whether to use an intercept
 * @param[in]  start_sum  At which index to start the sum
 * @param[in]  start_y    First used y index (observation)
 * @param[in]  start_v    First used v index (residual)
 */
template <typename DataT>
__global__ void sum_of_squares_kernel(const DataT* d_y, const DataT* d_mu,
                                      const DataT* d_ar, const DataT* d_ma,
                                      const DataT* d_sar, const DataT* d_sma,
                                      DataT* d_loglike, int n_obs, int n_phi,
                                      int n_theta, int p, int q, int P, int Q,
                                      int s, int k, int start_sum, int start_y,
                                      int start_v) {
  // Load phi, theta and mu to registers
  DataT phi, theta;
  if (threadIdx.x < n_phi) {
    phi = MLCommon::TimeSeries::reduced_polynomial<true>(
      blockIdx.x, d_ar, p, d_sar, P, s, threadIdx.x + 1);
  }
  if (threadIdx.x < n_theta) {
    theta = MLCommon::TimeSeries::reduced_polynomial<false>(
      blockIdx.x, d_ma, q, d_sma, Q, s, threadIdx.x + 1);
  }
  DataT mu = k ? d_mu[blockIdx.x] : (DataT)0;

  // Shared memory: load y and initialize the residuals
  extern __shared__ DataT shared_mem[];
  DataT* b_y = shared_mem;
  DataT* b_vs = shared_mem + n_obs - start_y;
  for (int i = threadIdx.x; i < n_obs - start_y; i += blockDim.x) {
    b_y[i] = d_y[n_obs * blockIdx.x + i + start_y];
  }
  for (int i = threadIdx.x; i < start_sum - start_v; i += blockDim.x) {
    b_vs[i] = (DataT)0;
  }

  // Main loop
  char* temp_smem = (char*)(shared_mem + 2 * n_obs - start_y - start_v);
  DataT res, ssq = 0;
  for (int i = start_sum; i < n_obs; i++) {
    __syncthreads();
    res = (DataT)0;
    res -=
      threadIdx.x < n_phi ? phi * b_y[i - threadIdx.x - 1 - start_y] : (DataT)0;
    res -= threadIdx.x < n_theta ? theta * b_vs[i - threadIdx.x - 1 - start_v]
                                 : (DataT)0;
    res = MLCommon::blockReduce(res, temp_smem);
    if (threadIdx.x == 0) {
      res += b_y[i - start_y] - mu;
      b_vs[i - start_v] = res;
      ssq += res * res;
    }
  }

  // Compute log-likelihood and write it to global memory
  if (threadIdx.x == 0) {
    d_loglike[blockIdx.x] =
      -0.5 * static_cast<DataT>(n_obs) *
      MLCommon::myLog(ssq / static_cast<DataT>(n_obs - start_sum));
  }
}

/**
 * Sum-of-squares estimation method
 *
 * @param[in]  handle     cuML handle
 * @param[in]  d_y        Series to fit: shape = (n_obs, batch_size)
 * @param[in]  batch_size Number of time series
 * @param[in]  n_obs      Number of observations in a time series
 * @param[in]  order      ARIMA hyper-parameters
 * @param[in]  Tparams    Transformed parameters
 * @param[out] d_loglike  Evaluated log-likelihood (device)
 * @param[in]  truncate   Number of observations to skip in the sum
 */
void conditional_sum_of_squares(cumlHandle& handle, const double* d_y,
                                int batch_size, int n_obs,
                                const ARIMAOrder& order,
                                const ARIMAParams<double>& Tparams,
                                double* d_loglike, int truncate) {
  ML::PUSH_RANGE(__func__);
  auto stream = handle.getStream();

  int n_phi = order.n_phi();
  int n_theta = order.n_theta();
  int max_lags = std::max(n_phi, n_theta);
  int start_sum = std::max(max_lags, truncate);
  int start_y = start_sum - n_phi;
  int start_v = start_sum - n_theta;

  // Compute the sum-of-squares and the log-likelihood
  int n_warps = std::max(MLCommon::ceildiv<int>(max_lags, 32), 1);
  size_t shared_mem_size =
    (2 * n_obs - start_y - start_v + n_warps) * sizeof(double);
  sum_of_squares_kernel<<<batch_size, 32 * n_warps, shared_mem_size, stream>>>(
    d_y, Tparams.mu, Tparams.ar, Tparams.ma, Tparams.sar, Tparams.sma,
    d_loglike, n_obs, n_phi, n_theta, order.p, order.q, order.P, order.Q,
    order.s, order.k, start_sum, start_y, start_v);
  CUDA_CHECK(hipPeekAtLastError());

  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, const ARIMAOrder& order,
                     const ARIMAParams<double>& params, double* loglike,
                     double* d_vs, bool trans, bool host_loglike,
                     LoglikeMethod method, int truncate, int fc_steps,
                     double* d_fc) {
  ML::PUSH_RANGE(__func__);

  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  ARIMAParams<double> Tparams;

  if (method != MLE && fc_steps) {
    /// TODO: add warning when solving #2232
    method = MLE;
  }

  /* Create log-likelihood device array if host pointer is provided */
  double* d_loglike;
  MLCommon::device_buffer<double> loglike_buffer(allocator, stream);
  if (host_loglike) {
    loglike_buffer.resize(batch_size, stream);
    d_loglike = loglike_buffer.data();
  } else {
    d_loglike = loglike;
  }

  if (trans) {
    Tparams.allocate(order, batch_size, allocator, stream, true);

    MLCommon::TimeSeries::batched_jones_transform(
      order, batch_size, false, params, Tparams, allocator, stream);

    Tparams.mu = params.mu;
  } else {
    // non-transformed case: just use original parameters
    Tparams = params;
  }

  if (!order.need_prep()) {
    if (method == CSS) {
      conditional_sum_of_squares(handle, d_y, batch_size, n_obs, order, Tparams,
                                 d_loglike, truncate);
    } else {
      batched_kalman_filter(handle, d_y, n_obs, Tparams, order, batch_size,
                            d_loglike, d_vs, fc_steps, d_fc);
    }
  } else {
    MLCommon::device_buffer<double> y_prep(
      allocator, stream, batch_size * (n_obs - order.lost_in_diff()));
    double* d_y_prep = y_prep.data();

    MLCommon::TimeSeries::prepare_data(d_y_prep, d_y, batch_size, n_obs,
                                       order.d, order.D, order.s, stream);

    if (method == CSS) {
      conditional_sum_of_squares(handle, d_y_prep, batch_size,
                                 n_obs - order.lost_in_diff(), order, Tparams,
                                 d_loglike, truncate);
    } else {
      batched_kalman_filter(handle, d_y_prep, n_obs - order.lost_in_diff(),
                            Tparams, order, batch_size, d_loglike, d_vs,
                            fc_steps, d_fc);
    }
  }

  if (host_loglike) {
    /* Tranfer log-likelihood device -> host */
    MLCommon::updateHost(loglike, d_loglike, batch_size, stream);
  }

  if (trans) {
    Tparams.deallocate(order, batch_size, allocator, stream, true);
  }
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, const ARIMAOrder& order, const double* d_params,
                     double* loglike, double* d_vs, bool trans,
                     bool host_loglike, LoglikeMethod method, int truncate,
                     int fc_steps, double* d_fc) {
  ML::PUSH_RANGE(__func__);

  // unpack parameters
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  ARIMAParams<double> params;
  params.allocate(order, batch_size, allocator, stream, false);
  params.unpack(order, batch_size, d_params, stream);

  batched_loglike(handle, d_y, batch_size, n_obs, order, params, loglike, d_vs,
                  trans, host_loglike, method, truncate, fc_steps, d_fc);

  params.deallocate(order, batch_size, allocator, stream, false);
  ML::POP_RANGE();
}

void batched_loglike_grad(cumlHandle& handle, const double* d_y, int batch_size,
                          int n_obs, const ARIMAOrder& order, const double* d_x,
                          double* d_grad, double h, bool trans,
                          LoglikeMethod method, int truncate) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  auto counting = thrust::make_counting_iterator(0);
  int N = order.complexity();

  // Initialize the perturbed x vector
  MLCommon::device_buffer<double> x_pert(allocator, stream, N * batch_size);
  double* d_x_pert = x_pert.data();
  MLCommon::copy(d_x_pert, d_x, N * batch_size, stream);

  // Create buffers for the log-likelihood and residuals
  MLCommon::device_buffer<double> ll_pos(allocator, stream, batch_size);
  MLCommon::device_buffer<double> ll_neg(allocator, stream, batch_size);
  MLCommon::device_buffer<double> res(
    allocator, stream, (n_obs - order.lost_in_diff()) * batch_size);
  double* d_ll_pos = ll_pos.data();
  double* d_ll_neg = ll_neg.data();

  for (int i = 0; i < N; i++) {
    // Add the perturbation to the i-th parameter
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_x_pert[N * bid + i] = d_x[N * bid + i] + h;
                     });

    // Evaluate the log-likelihood with the positive perturbation
    batched_loglike(handle, d_y, batch_size, n_obs, order, d_x_pert, d_ll_pos,
                    res.data(), trans, false, method, truncate);

    // Subtract the perturbation to the i-th parameter
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_x_pert[N * bid + i] = d_x[N * bid + i] - h;
                     });

    // Evaluate the log-likelihood with the negative perturbation
    batched_loglike(handle, d_y, batch_size, n_obs, order, d_x_pert, d_ll_neg,
                    res.data(), trans, false, method, truncate);

    // First derivative with a second-order accuracy
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_grad[N * bid + i] =
                         (d_ll_pos[bid] - d_ll_neg[bid]) / (2.0 * h);
                     });

    // Reset the i-th parameter
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size,
      [=] __device__(int bid) { d_x_pert[N * bid + i] = d_x[N * bid + i]; });
  }

  ML::POP_RANGE();
}

void information_criterion(cumlHandle& handle, const double* d_y,
                           int batch_size, int n_obs, const ARIMAOrder& order,
                           const ARIMAParams<double>& params, double* d_ic,
                           int ic_type) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double* d_vs = (double*)allocator->allocate(
    sizeof(double) * (n_obs - order.lost_in_diff()) * batch_size, stream);

  /* Compute log-likelihood in d_ic */
  batched_loglike(handle, d_y, batch_size, n_obs, order, params, d_ic, d_vs,
                  false, false);

  /* Compute information criterion from log-likelihood and base term */
  MLCommon::Metrics::Batched::information_criterion(
    d_ic, d_ic, static_cast<MLCommon::Metrics::IC_Type>(ic_type),
    order.complexity(), batch_size, n_obs - order.lost_in_diff(), stream);

  allocator->deallocate(
    d_vs, sizeof(double) * (n_obs - order.lost_in_diff()) * batch_size, stream);
  ML::POP_RANGE();
}

/**
 * Test that the parameters are valid for the inverse transform
 * 
 * @tparam isAr        Are these (S)AR or (S)MA parameters?
 * @param[in]  params  Parameters
 * @param[in]  pq      p for AR, q for MA, P for SAR, Q for SMA
 */
template <bool isAr>
DI bool test_invparams(const double* params, int pq) {
  double new_params[4];
  double tmp[4];

  constexpr double coef = isAr ? 1 : -1;

  for (int i = 0; i < pq; i++) {
    tmp[i] = params[i];
    new_params[i] = tmp[i];
  }

  // Perform inverse transform and stop before atanh step
  for (int j = pq - 1; j > 0; --j) {
    double a = new_params[j];
    for (int k = 0; k < j; ++k) {
      tmp[k] =
        (new_params[k] + coef * a * new_params[j - k - 1]) / (1 - (a * a));
    }
    for (int iter = 0; iter < j; ++iter) {
      new_params[iter] = tmp[iter];
    }
  }

  // Verify that the values are between -1 and 1
  bool result = true;
  for (int i = 0; i < pq; i++) {
    result = result && !(new_params[i] <= -1 || new_params[i] >= 1);
  }
  return result;
}

/**
 * Auxiliary function of _start_params: least square approximation of an
 * ARMA model (with or without seasonality)
 * @note: in this function the non-seasonal case has s=1, not s=0!
 */
void _arma_least_squares(cumlHandle& handle, double* d_ar, double* d_ma,
                         double* d_sigma2,
                         const MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                         int p, int q, int s, bool estimate_sigma2, int k = 0,
                         double* d_mu = nullptr) {
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();
  auto counting = thrust::make_counting_iterator(0);

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  int ps = p * s, qs = q * s;
  int p_ar = std::max(ps, 2 * qs);
  int r = std::max(p_ar + qs, ps);

  if ((q && p_ar >= n_obs - p_ar) || p + q + k >= n_obs - r) {
    // Too few observations for the estimate, fill with 0 (1 for sigma2)
    if (k)
      CUDA_CHECK(hipMemsetAsync(d_mu, 0, sizeof(double) * batch_size, stream));
    if (p)
      CUDA_CHECK(
        hipMemsetAsync(d_ar, 0, sizeof(double) * p * batch_size, stream));
    if (q)
      CUDA_CHECK(
        hipMemsetAsync(d_ma, 0, sizeof(double) * q * batch_size, stream));
    if (estimate_sigma2) {
      thrust::device_ptr<double> sigma2_thrust =
        thrust::device_pointer_cast(d_sigma2);
      thrust::fill(thrust::cuda::par.on(stream), sigma2_thrust,
                   sigma2_thrust + batch_size, 1.0);
    }
    return;
  }

  /* Matrix formed by lag matrices of y and the residuals respectively,
   * side by side. The left side will be used to estimate AR, the right
   * side to estimate MA */
  MLCommon::LinAlg::Batched::Matrix<double> bm_ls_ar_res(
    n_obs - r, p + q + k, batch_size, cublas_handle, allocator, stream, false);
  int ar_offset = r - ps;
  int res_offset = r - p_ar - qs;

  // Get residuals from an AR(p_ar) model to estimate the MA parameters
  if (q) {
    // Create lagged y
    int ls_height = n_obs - p_ar;
    MLCommon::LinAlg::Batched::Matrix<double> bm_ls =
      MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, p_ar);

    /* Matrix for the initial AR fit, initialized by copy of y
     * (note: this is because gels works in-place ; the matrix has larger
     *  dimensions than the actual AR fit) */
    MLCommon::LinAlg::Batched::Matrix<double> bm_ar_fit =
      MLCommon::LinAlg::Batched::b_2dcopy(bm_y, p_ar, 0, ls_height, 1);

    // Residual, initialized as offset y to avoid one kernel call
    MLCommon::LinAlg::Batched::Matrix<double> bm_residual(bm_ar_fit);

    // Initial AR fit
    MLCommon::LinAlg::Batched::b_gels(bm_ls, bm_ar_fit);

    // Compute residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, ls_height, 1, p_ar, -1.0,
                                      bm_ls, bm_ar_fit, 1.0, bm_residual);

    // Lags of the residual
    MLCommon::LinAlg::Batched::b_lagged_mat(bm_residual, bm_ls_ar_res, q,
                                            n_obs - r, res_offset,
                                            (n_obs - r) * (k + p), s);
  }

  // Fill the first column of the matrix with 1 if we fit an intercept
  if (k) {
    double* d_ls_ar_res = bm_ls_ar_res.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double* b_ls_ar_res =
                         d_ls_ar_res + bid * (n_obs - r) * (p + q + k);
                       for (int i = 0; i < n_obs - r; i++) {
                         b_ls_ar_res[i] = 1.0;
                       }
                     });
  }

  // Lags of y
  MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, bm_ls_ar_res, p, n_obs - r,
                                          ar_offset, (n_obs - r) * k, s);

  /* Initializing the vector for the ARMA fit
   * (note: also in-place as described for AR fit) */
  MLCommon::LinAlg::Batched::Matrix<double> bm_arma_fit =
    MLCommon::LinAlg::Batched::b_2dcopy(bm_y, r, 0, n_obs - r, 1);

  // The residuals will be computed only if sigma2 is requested
  MLCommon::LinAlg::Batched::Matrix<double> bm_final_residual(
    n_obs - r, 1, batch_size, cublas_handle, allocator, stream, false);
  if (estimate_sigma2) {
    MLCommon::copy(bm_final_residual.raw_data(), bm_arma_fit.raw_data(),
                   (n_obs - r) * batch_size, stream);
  }

  // ARMA fit
  MLCommon::LinAlg::Batched::b_gels(bm_ls_ar_res, bm_arma_fit);

  // Copy the results in the parameter vectors
  const double* d_arma_fit = bm_arma_fit.raw_data();
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     const double* b_arma_fit = d_arma_fit + bid * (n_obs - r);
                     if (k) {
                       d_mu[bid] = b_arma_fit[0];
                     }
                     if (p) {
                       double* b_ar = d_ar + bid * p;
                       for (int i = 0; i < p; i++) {
                         b_ar[i] = b_arma_fit[i + k];
                       }
                     }
                     if (q) {
                       double* b_ma = d_ma + bid * q;
                       for (int i = 0; i < q; i++) {
                         b_ma[i] = b_arma_fit[i + p + k];
                       }
                     }
                   });

  if (estimate_sigma2) {
    // Compute final residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, n_obs - r, 1, p + q + k,
                                      -1.0, bm_ls_ar_res, bm_arma_fit, 1.0,
                                      bm_final_residual);

    // Compute variance
    double* d_residual = bm_final_residual.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double acc = 0.0;
                       const double* b_residual =
                         d_residual + (n_obs - r) * bid;
                       for (int i = q; i < n_obs - r; i++) {
                         double res = b_residual[i];
                         acc += res * res;
                       }
                       d_sigma2[bid] = acc / static_cast<double>(n_obs - r - q);
                     });
  }

  // If (S)AR or (S)MA are not valid for the inverse transform, set them to zero
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     if (p) {
                       double* b_ar = d_ar + bid * p;
                       bool valid = test_invparams<true>(b_ar, p);
                       if (!valid) {
                         for (int ip = 0; ip < p; ip++) b_ar[ip] = 0;
                       }
                     }
                     if (q) {
                       double* b_ma = d_ma + bid * q;
                       bool valid = test_invparams<false>(b_ma, q);
                       if (!valid) {
                         for (int iq = 0; iq < q; iq++) b_ma[iq] = 0;
                       }
                     }
                   });
}

/**
 * Auxiliary function of estimate_x0: compute the starting parameters for
 * the series pre-processed by estimate_x0
 */
void _start_params(cumlHandle& handle, ARIMAParams<double>& params,
                   const MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                   const ARIMAOrder& order) {
  // Estimate an ARMA fit without seasonality
  if (order.p + order.q + order.k)
    _arma_least_squares(handle, params.ar, params.ma, params.sigma2, bm_y,
                        order.p, order.q, 1, true, order.k, params.mu);

  // Estimate a seasonal ARMA fit independantly
  if (order.P + order.Q)
    _arma_least_squares(handle, params.sar, params.sma, params.sigma2, bm_y,
                        order.P, order.Q, order.s,
                        order.p + order.q + order.k == 0);
}

void estimate_x0(cumlHandle& handle, ARIMAParams<double>& params,
                 const double* d_y, int batch_size, int n_obs,
                 const ARIMAOrder& order) {
  ML::PUSH_RANGE(__func__);
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  // Difference if necessary, copy otherwise
  MLCommon::LinAlg::Batched::Matrix<double> bm_yd(
    n_obs - order.d - order.s * order.D, 1, batch_size, cublas_handle,
    allocator, stream, false);
  MLCommon::TimeSeries::prepare_data(bm_yd.raw_data(), d_y, batch_size, n_obs,
                                     order.d, order.D, order.s, stream);

  // Do the computation of the initial parameters
  _start_params(handle, params, bm_yd, order);
  ML::POP_RANGE();
}

}  // namespace ML
