#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <cuml/cuml.hpp>
#include <cuml/tsa/batched_arima.hpp>
#include <cuml/tsa/batched_kalman.hpp>

#include "common/cumlHandle.hpp"
#include "common/nvtx.hpp"
#include "cuda_utils.h"
#include "linalg/batched/matrix.h"
#include "linalg/matrix_vector_op.h"
#include "metrics/batched/information_criterion.h"
#include "timeSeries/arima_helpers.h"
#include "utils.h"

namespace ML {

void predict(cumlHandle& handle, const double* d_y, int batch_size, int n_obs,
             int start, int end, const ARIMAOrder& order,
             const ARIMAParams<double>& params, double* d_vs, double* d_y_p) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  const auto stream = handle.getStream();

  // Prepare data
  int diff_obs = order.lost_in_diff();
  int ld_yprep = n_obs - diff_obs;
  double* d_y_prep = (double*)allocator->allocate(
    ld_yprep * batch_size * sizeof(double), stream);
  MLCommon::TimeSeries::prepare_data(d_y_prep, d_y, batch_size, n_obs, order.d,
                                     order.D, order.s, stream, order.k,
                                     params.mu);

  // Create temporary array for the forecasts
  int num_steps = std::max(end - n_obs, 0);
  double* d_y_fc = nullptr;
  if (num_steps) {
    d_y_fc = (double*)allocator->allocate(
      num_steps * batch_size * sizeof(double), stream);
  }

  // Compute the residual and forecast - provide already prepared data and
  // extracted parameters
  ARIMAOrder order_after_prep = {order.p, 0,       order.q, order.P,
                                 0,       order.Q, order.s, 0};
  std::vector<double> loglike = std::vector<double>(batch_size);
  batched_loglike(handle, d_y_prep, batch_size, n_obs - diff_obs,
                  order_after_prep, params, loglike.data(), d_vs, false, true,
                  num_steps, d_y_fc);

  auto counting = thrust::make_counting_iterator(0);
  int predict_ld = end - start;

  //
  // In-sample prediction
  //

  int p_start = std::max(start, diff_obs);
  int p_end = std::min(n_obs, end);

  // The prediction loop starts by filling undefined predictions with NaN,
  // then computes the predictions from the observations and residuals
  if (start < n_obs) {
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_y_p[0] = 0.0;
                       for (int i = 0; i < diff_obs - start; i++) {
                         d_y_p[bid * predict_ld + i] = nan("");
                       }
                       for (int i = p_start; i < p_end; i++) {
                         d_y_p[bid * predict_ld + i - start] =
                           d_y[bid * n_obs + i] -
                           d_vs[bid * ld_yprep + i - diff_obs];
                       }
                     });
  }

  //
  // Finalize out-of-sample forecast and copy in-sample predictions
  //

  if (num_steps) {
    // Add trend and/or undiff
    MLCommon::TimeSeries::finalize_forecast(
      d_y_fc, d_y, num_steps, batch_size, n_obs, n_obs, order.d, order.D,
      order.s, stream, order.k, params.mu);

    // Copy forecast in d_y_p
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       for (int i = 0; i < num_steps; i++) {
                         d_y_p[bid * predict_ld + n_obs - start + i] =
                           d_y_fc[num_steps * bid + i];
                       }
                     });

    allocator->deallocate(d_y_fc, num_steps * batch_size * sizeof(double),
                          stream);
  }

  allocator->deallocate(d_y_prep, ld_yprep * batch_size * sizeof(double),
                        stream);
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, const ARIMAOrder& order,
                     const ARIMAParams<double>& params, double* loglike,
                     double* d_vs, bool trans, bool host_loglike, int fc_steps,
                     double* d_fc) {
  ML::PUSH_RANGE(__func__);

  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  ARIMAParams<double> Tparams;

  if (trans) {
    Tparams.allocate(order, batch_size, allocator, stream, true);

    MLCommon::TimeSeries::batched_jones_transform(
      order, batch_size, false, params, Tparams, allocator, stream);
  } else {
    // non-transformed case: just use original parameters
    Tparams = params;
  }

  if (!order.need_prep()) {
    batched_kalman_filter(handle, d_y, n_obs, Tparams, order, batch_size,
                          loglike, d_vs, host_loglike, fc_steps, d_fc);
  } else {
    double* d_y_prep = (double*)allocator->allocate(
      batch_size * (n_obs - order.d - order.s * order.D) * sizeof(double),
      stream);

    MLCommon::TimeSeries::prepare_data(d_y_prep, d_y, batch_size, n_obs,
                                       order.d, order.D, order.s, stream,
                                       order.k, params.mu);

    batched_kalman_filter(handle, d_y_prep, n_obs - order.d - order.s * order.D,
                          Tparams, order, batch_size, loglike, d_vs,
                          host_loglike, fc_steps, d_fc);

    allocator->deallocate(
      d_y_prep,
      sizeof(double) * batch_size * (n_obs - order.d - order.s * order.D),
      stream);
  }

  if (trans) {
    Tparams.deallocate(order, batch_size, allocator, stream, true);
  }
  ML::POP_RANGE();
}

void batched_loglike(cumlHandle& handle, const double* d_y, int batch_size,
                     int n_obs, const ARIMAOrder& order, const double* d_params,
                     double* loglike, double* d_vs, bool trans,
                     bool host_loglike, int fc_steps, double* d_fc) {
  ML::PUSH_RANGE(__func__);

  // unpack parameters
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  ARIMAParams<double> params;
  params.allocate(order, batch_size, allocator, stream, false);
  params.unpack(order, batch_size, d_params, stream);

  batched_loglike(handle, d_y, batch_size, n_obs, order, params, loglike, d_vs,
                  trans, host_loglike, fc_steps, d_fc);

  params.deallocate(order, batch_size, allocator, stream, false);
  ML::POP_RANGE();
}

void information_criterion(cumlHandle& handle, const double* d_y,
                           int batch_size, int n_obs, const ARIMAOrder& order,
                           const ARIMAParams<double>& params, double* ic,
                           int ic_type) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.getDeviceAllocator();
  auto stream = handle.getStream();
  double* d_vs = (double*)allocator->allocate(
    sizeof(double) * (n_obs - order.lost_in_diff()) * batch_size, stream);
  double* d_ic =
    (double*)allocator->allocate(sizeof(double) * batch_size, stream);

  /* Compute log-likelihood in d_ic */
  batched_loglike(handle, d_y, batch_size, n_obs, order, params, d_ic, d_vs,
                  false, false);

  /* Compute information criterion from log-likelihood and base term */
  MLCommon::Metrics::Batched::information_criterion(
    d_ic, d_ic, static_cast<MLCommon::Metrics::IC_Type>(ic_type),
    order.complexity(), batch_size, n_obs - order.lost_in_diff(), stream);

  /* Transfer information criterion device -> host */
  MLCommon::updateHost(ic, d_ic, batch_size, stream);

  allocator->deallocate(
    d_vs, sizeof(double) * (n_obs - order.lost_in_diff()) * batch_size, stream);
  allocator->deallocate(d_ic, sizeof(double) * batch_size, stream);
  ML::POP_RANGE();
}

/**
 * Test that the parameters are valid for the inverse transform
 * 
 * @tparam isAr        Are these (S)AR or (S)MA parameters?
 * @param[in]  params  Parameters
 * @param[in]  pq      p for AR, q for MA, P for SAR, Q for SMA
 */
template <bool isAr>
DI bool test_invparams(const double* params, int pq) {
  double new_params[4];
  double tmp[4];

  constexpr double coef = isAr ? 1 : -1;

  for (int i = 0; i < pq; i++) {
    tmp[i] = params[i];
    new_params[i] = tmp[i];
  }

  // Perform inverse transform and stop before atanh step
  for (int j = pq - 1; j > 0; --j) {
    double a = new_params[j];
    for (int k = 0; k < j; ++k) {
      tmp[k] =
        (new_params[k] + coef * a * new_params[j - k - 1]) / (1 - (a * a));
    }
    for (int iter = 0; iter < j; ++iter) {
      new_params[iter] = tmp[iter];
    }
  }

  // Verify that the values are between -1 and 1
  bool result = true;
  for (int i = 0; i < pq; i++) {
    result = result && !(new_params[i] <= -1 || new_params[i] >= 1);
  }
  return result;
}

/**
 * Auxiliary function of _start_params: least square approximation of an
 * ARMA model (with or without seasonality)
 * @note: in this function the non-seasonal case has s=1, not s=0!
 */
void _arma_least_squares(cumlHandle& handle, double* d_ar, double* d_ma,
                         double* d_sigma2,
                         const MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                         int p, int q, int s, bool estimate_sigma2, int k = 0,
                         double* d_mu = nullptr) {
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();
  auto counting = thrust::make_counting_iterator(0);

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  int ps = p * s, qs = q * s;
  int p_ar = std::max(ps, 2 * qs);
  int r = std::max(p_ar + qs, ps);

  if ((q && p_ar >= n_obs - p_ar) || p + q + k >= n_obs - r) {
    // Too few observations for the estimate, fill with 0 (1 for sigma2)
    if (k)
      CUDA_CHECK(hipMemsetAsync(d_mu, 0, sizeof(double) * batch_size, stream));
    if (p)
      CUDA_CHECK(
        hipMemsetAsync(d_ar, 0, sizeof(double) * p * batch_size, stream));
    if (q)
      CUDA_CHECK(
        hipMemsetAsync(d_ma, 0, sizeof(double) * q * batch_size, stream));
    if (estimate_sigma2) {
      thrust::device_ptr<double> sigma2_thrust =
        thrust::device_pointer_cast(d_sigma2);
      thrust::fill(thrust::cuda::par.on(stream), sigma2_thrust,
                   sigma2_thrust + batch_size, 1.0);
    }
    return;
  }

  /* Matrix formed by lag matrices of y and the residuals respectively,
   * side by side. The left side will be used to estimate AR, the right
   * side to estimate MA */
  MLCommon::LinAlg::Batched::Matrix<double> bm_ls_ar_res(
    n_obs - r, p + q + k, batch_size, cublas_handle, allocator, stream, false);
  int ar_offset = r - ps;
  int res_offset = r - p_ar - qs;

  // Get residuals from an AR(p_ar) model to estimate the MA parameters
  if (q) {
    // Create lagged y
    int ls_height = n_obs - p_ar;
    MLCommon::LinAlg::Batched::Matrix<double> bm_ls =
      MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, p_ar);

    /* Matrix for the initial AR fit, initialized by copy of y
     * (note: this is because gels works in-place ; the matrix has larger
     *  dimensions than the actual AR fit) */
    MLCommon::LinAlg::Batched::Matrix<double> bm_ar_fit =
      MLCommon::LinAlg::Batched::b_2dcopy(bm_y, p_ar, 0, ls_height, 1);

    // Residual, initialized as offset y to avoid one kernel call
    MLCommon::LinAlg::Batched::Matrix<double> bm_residual(bm_ar_fit);

    // Initial AR fit
    MLCommon::LinAlg::Batched::b_gels(bm_ls, bm_ar_fit);

    // Compute residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, ls_height, 1, p_ar, -1.0,
                                      bm_ls, bm_ar_fit, 1.0, bm_residual);

    // Lags of the residual
    MLCommon::LinAlg::Batched::b_lagged_mat(bm_residual, bm_ls_ar_res, q,
                                            n_obs - r, res_offset,
                                            (n_obs - r) * (k + p), s);
  }

  // Fill the first column of the matrix with 1 if we fit an intercept
  if (k) {
    double* d_ls_ar_res = bm_ls_ar_res.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double* b_ls_ar_res =
                         d_ls_ar_res + bid * (n_obs - r) * (p + q + k);
                       for (int i = 0; i < n_obs - r; i++) {
                         b_ls_ar_res[i] = 1.0;
                       }
                     });
  }

  // Lags of y
  MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, bm_ls_ar_res, p, n_obs - r,
                                          ar_offset, (n_obs - r) * k, s);

  /* Initializing the vector for the ARMA fit
   * (note: also in-place as described for AR fit) */
  MLCommon::LinAlg::Batched::Matrix<double> bm_arma_fit =
    MLCommon::LinAlg::Batched::b_2dcopy(bm_y, r, 0, n_obs - r, 1);

  // The residuals will be computed only if sigma2 is requested
  MLCommon::LinAlg::Batched::Matrix<double> bm_final_residual(
    n_obs - r, 1, batch_size, cublas_handle, allocator, stream, false);
  if (estimate_sigma2) {
    MLCommon::copy(bm_final_residual.raw_data(), bm_arma_fit.raw_data(),
                   (n_obs - r) * batch_size, stream);
  }

  // ARMA fit
  MLCommon::LinAlg::Batched::b_gels(bm_ls_ar_res, bm_arma_fit);

  // Copy the results in the parameter vectors
  const double* d_arma_fit = bm_arma_fit.raw_data();
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     const double* b_arma_fit = d_arma_fit + bid * (n_obs - r);
                     if (k) {
                       d_mu[bid] = b_arma_fit[0];
                     }
                     if (p) {
                       double* b_ar = d_ar + bid * p;
                       for (int i = 0; i < p; i++) {
                         b_ar[i] = b_arma_fit[i + k];
                       }
                     }
                     if (q) {
                       double* b_ma = d_ma + bid * q;
                       for (int i = 0; i < q; i++) {
                         b_ma[i] = b_arma_fit[i + p + k];
                       }
                     }
                   });

  if (estimate_sigma2) {
    // Compute final residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, n_obs - r, 1, p + q + k,
                                      -1.0, bm_ls_ar_res, bm_arma_fit, 1.0,
                                      bm_final_residual);

    // Compute variance
    double* d_residual = bm_final_residual.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double acc = 0.0;
                       const double* b_residual =
                         d_residual + (n_obs - r) * bid;
                       for (int i = q; i < n_obs - r; i++) {
                         double res = b_residual[i];
                         acc += res * res;
                       }
                       d_sigma2[bid] = acc / static_cast<double>(n_obs - r - q);
                     });
  }

  // If (S)AR or (S)MA are not valid for the inverse transform, set them to zero
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     if (p) {
                       double* b_ar = d_ar + bid * p;
                       bool valid = test_invparams<true>(b_ar, p);
                       if (!valid) {
                         for (int ip = 0; ip < p; ip++) b_ar[ip] = 0;
                       }
                     }
                     if (q) {
                       double* b_ma = d_ma + bid * q;
                       bool valid = test_invparams<false>(b_ma, q);
                       if (!valid) {
                         for (int iq = 0; iq < q; iq++) b_ma[iq] = 0;
                       }
                     }
                   });
}

/**
 * Auxiliary function of estimate_x0: compute the starting parameters for
 * the series pre-processed by estimate_x0
 */
void _start_params(cumlHandle& handle, ARIMAParams<double>& params,
                   const MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                   const ARIMAOrder& order) {
  // Estimate an ARMA fit without seasonality
  if (order.p + order.q + order.k)
    _arma_least_squares(handle, params.ar, params.ma, params.sigma2, bm_y,
                        order.p, order.q, 1, true, order.k, params.mu);

  // Estimate a seasonal ARMA fit independantly
  if (order.P + order.Q)
    _arma_least_squares(handle, params.sar, params.sma, params.sigma2, bm_y,
                        order.P, order.Q, order.s,
                        order.p + order.q + order.k == 0);
}

void estimate_x0(cumlHandle& handle, ARIMAParams<double>& params,
                 const double* d_y, int batch_size, int n_obs,
                 const ARIMAOrder& order) {
  ML::PUSH_RANGE(__func__);
  const auto& handle_impl = handle.getImpl();
  auto stream = handle_impl.getStream();
  auto cublas_handle = handle_impl.getCublasHandle();
  auto allocator = handle_impl.getDeviceAllocator();

  // Difference if necessary, copy otherwise
  MLCommon::LinAlg::Batched::Matrix<double> bm_yd(
    n_obs - order.d - order.s * order.D, 1, batch_size, cublas_handle,
    allocator, stream, false);
  MLCommon::TimeSeries::prepare_data(bm_yd.raw_data(), d_y, batch_size, n_obs,
                                     order.d, order.D, order.s, stream);
  // Note: mu is not known yet! We just want to difference the data

  // Do the computation of the initial parameters
  _start_params(handle, params, bm_yd, order);
  ML::POP_RANGE();
}

}  // namespace ML
