#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <cuml/cuml.hpp>
#include <cuml/tsa/batched_arima.hpp>
#include <cuml/tsa/batched_kalman.hpp>

#include <raft/cudart_utils.h>
#include <common/nvtx.hpp>
#include <cuml/common/device_buffer.hpp>
#include <linalg/batched/matrix.cuh>
#include <metrics/batched/information_criterion.cuh>
#include <raft/cuda_utils.cuh>
#include <raft/linalg/matrix_vector_op.cuh>
#include <timeSeries/arima_helpers.cuh>

namespace ML {

void pack(raft::handle_t& handle, const ARIMAParams<double>& params,
          const ARIMAOrder& order, int batch_size, double* param_vec) {
  const auto stream = handle.get_stream();
  params.pack(order, batch_size, param_vec, stream);
}

void unpack(raft::handle_t& handle, ARIMAParams<double>& params,
            const ARIMAOrder& order, int batch_size, const double* param_vec) {
  const auto stream = handle.get_stream();
  params.unpack(order, batch_size, param_vec, stream);
}

void batched_diff(raft::handle_t& handle, double* d_y_diff, const double* d_y,
                  int batch_size, int n_obs, const ARIMAOrder& order) {
  const auto stream = handle.get_stream();
  MLCommon::TimeSeries::prepare_data(d_y_diff, d_y, batch_size, n_obs, order.d,
                                     order.D, order.s, stream);
}

void predict(raft::handle_t& handle, const double* d_y, int batch_size,
             int n_obs, int start, int end, const ARIMAOrder& order,
             const ARIMAParams<double>& params, double* d_y_p, bool pre_diff,
             double level, double* d_lower, double* d_upper) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.get_device_allocator();
  const auto stream = handle.get_stream();

  bool diff = order.need_diff() && pre_diff && level == 0;

  // Prepare data
  int n_obs_kf;
  const double* d_y_kf;
  MLCommon::device_buffer<double> diff_buffer(allocator, stream);
  ARIMAOrder order_after_prep = order;
  if (diff) {
    n_obs_kf = n_obs - order.n_diff();
    diff_buffer.resize(n_obs_kf * batch_size, stream);
    MLCommon::TimeSeries::prepare_data(diff_buffer.data(), d_y, batch_size,
                                       n_obs, order.d, order.D, order.s,
                                       stream);
    d_y_kf = diff_buffer.data();
    order_after_prep.d = 0;
    order_after_prep.D = 0;
  } else {
    n_obs_kf = n_obs;
    d_y_kf = d_y;
  }

  // Create temporary array for the residuals
  MLCommon::device_buffer<double> v_buffer(allocator, stream,
                                           n_obs_kf * batch_size);
  double* d_vs = v_buffer.data();

  // Create temporary array for the forecasts
  int num_steps = std::max(end - n_obs, 0);
  MLCommon::device_buffer<double> fc_buffer(allocator, stream,
                                            num_steps * batch_size);
  double* d_y_fc = fc_buffer.data();

  // Compute the residual and forecast
  std::vector<double> loglike = std::vector<double>(batch_size);
  /// TODO: use device loglike to avoid useless copy ; part of #2233
  batched_loglike(handle, d_y_kf, batch_size, n_obs_kf, order_after_prep,
                  params, loglike.data(), d_vs, false, true, MLE, 0, num_steps,
                  d_y_fc, level, d_lower, d_upper);

  auto counting = thrust::make_counting_iterator(0);
  int predict_ld = end - start;

  //
  // In-sample prediction
  //

  int res_offset = diff ? order.d + order.s * order.D : 0;
  int p_start = std::max(start, res_offset);
  int p_end = std::min(n_obs, end);

  // The prediction loop starts by filling undefined predictions with NaN,
  // then computes the predictions from the observations and residuals
  if (start < n_obs) {
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_y_p[0] = 0.0;
                       for (int i = 0; i < res_offset - start; i++) {
                         d_y_p[bid * predict_ld + i] = nan("");
                       }
                       for (int i = p_start; i < p_end; i++) {
                         d_y_p[bid * predict_ld + i - start] =
                           d_y[bid * n_obs + i] -
                           d_vs[bid * n_obs_kf + i - res_offset];
                       }
                     });
  }

  //
  // Finalize out-of-sample forecast and copy in-sample predictions
  //

  if (num_steps) {
    if (diff) {
      MLCommon::TimeSeries::finalize_forecast(d_y_fc, d_y, num_steps,
                                              batch_size, n_obs, n_obs, order.d,
                                              order.D, order.s, stream);
    }

    // Copy forecast in d_y_p
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       for (int i = 0; i < num_steps; i++) {
                         d_y_p[bid * predict_ld + n_obs - start + i] =
                           d_y_fc[num_steps * bid + i];
                       }
                     });
    /// TODO: 2D copy kernel?
  }

  ML::POP_RANGE();
}

/**
 * Kernel to compute the sum-of-squares log-likelihood estimation
 *
 * @param[in]  d_y        Series to fit
 * @param[in]  d_mu       mu parameters
 * @param[in]  d_ar       AR parameters
 * @param[in]  d_ma       MA parameters
 * @param[in]  d_sar      Seasonal AR parameters
 * @param[in]  d_sma      Seasonal MA parameters
 * @param[out] d_loglike  Evaluated log-likelihood
 * @param[in]  n_obs      Number of observations in a time series
 * @param[in]  n_phi      Number of phi coefficients (combined AR-SAR)
 * @param[in]  n_theta    Number of theta coefficients (combined MA-SMA)
 * @param[in]  p          Number of AR parameters
 * @param[in]  q          Number of MA parameters
 * @param[in]  P          Number of seasonal AR parameters
 * @param[in]  Q          Number of seasonal MA parameters
 * @param[in]  s          Seasonal period or 0
 * @param[in]  k          Whether to use an intercept
 * @param[in]  start_sum  At which index to start the sum
 * @param[in]  start_y    First used y index (observation)
 * @param[in]  start_v    First used v index (residual)
 */
template <typename DataT>
__global__ void sum_of_squares_kernel(const DataT* d_y, const DataT* d_mu,
                                      const DataT* d_ar, const DataT* d_ma,
                                      const DataT* d_sar, const DataT* d_sma,
                                      DataT* d_loglike, int n_obs, int n_phi,
                                      int n_theta, int p, int q, int P, int Q,
                                      int s, int k, int start_sum, int start_y,
                                      int start_v) {
  // Load phi, theta and mu to registers
  DataT phi, theta;
  if (threadIdx.x < n_phi) {
    phi = MLCommon::TimeSeries::reduced_polynomial<true>(
      blockIdx.x, d_ar, p, d_sar, P, s, threadIdx.x + 1);
  }
  if (threadIdx.x < n_theta) {
    theta = MLCommon::TimeSeries::reduced_polynomial<false>(
      blockIdx.x, d_ma, q, d_sma, Q, s, threadIdx.x + 1);
  }
  DataT mu = k ? d_mu[blockIdx.x] : (DataT)0;

  // Shared memory: load y and initialize the residuals
  extern __shared__ DataT shared_mem[];
  DataT* b_y = shared_mem;
  DataT* b_vs = shared_mem + n_obs - start_y;
  for (int i = threadIdx.x; i < n_obs - start_y; i += blockDim.x) {
    b_y[i] = d_y[n_obs * blockIdx.x + i + start_y];
  }
  for (int i = threadIdx.x; i < start_sum - start_v; i += blockDim.x) {
    b_vs[i] = (DataT)0;
  }

  // Main loop
  char* temp_smem = (char*)(shared_mem + 2 * n_obs - start_y - start_v);
  DataT res, ssq = 0;
  for (int i = start_sum; i < n_obs; i++) {
    __syncthreads();
    res = (DataT)0;
    res -=
      threadIdx.x < n_phi ? phi * b_y[i - threadIdx.x - 1 - start_y] : (DataT)0;
    res -= threadIdx.x < n_theta ? theta * b_vs[i - threadIdx.x - 1 - start_v]
                                 : (DataT)0;
    res = raft::blockReduce(res, temp_smem);
    if (threadIdx.x == 0) {
      res += b_y[i - start_y] - mu;
      b_vs[i - start_v] = res;
      ssq += res * res;
    }
  }

  // Compute log-likelihood and write it to global memory
  if (threadIdx.x == 0) {
    d_loglike[blockIdx.x] =
      -0.5 * static_cast<DataT>(n_obs) *
      raft::myLog(ssq / static_cast<DataT>(n_obs - start_sum));
  }
}

/**
 * Sum-of-squares estimation method
 *
 * @param[in]  handle     cuML handle
 * @param[in]  d_y        Series to fit: shape = (n_obs, batch_size)
 * @param[in]  batch_size Number of time series
 * @param[in]  n_obs      Number of observations in a time series
 * @param[in]  order      ARIMA hyper-parameters
 * @param[in]  Tparams    Transformed parameters
 * @param[out] d_loglike  Evaluated log-likelihood (device)
 * @param[in]  truncate   Number of observations to skip in the sum
 */
void conditional_sum_of_squares(raft::handle_t& handle, const double* d_y,
                                int batch_size, int n_obs,
                                const ARIMAOrder& order,
                                const ARIMAParams<double>& Tparams,
                                double* d_loglike, int truncate) {
  ML::PUSH_RANGE(__func__);
  auto stream = handle.get_stream();

  int n_phi = order.n_phi();
  int n_theta = order.n_theta();
  int max_lags = std::max(n_phi, n_theta);
  int start_sum = std::max(max_lags, truncate);
  int start_y = start_sum - n_phi;
  int start_v = start_sum - n_theta;

  // Compute the sum-of-squares and the log-likelihood
  int n_warps = std::max(raft::ceildiv<int>(max_lags, 32), 1);
  size_t shared_mem_size =
    (2 * n_obs - start_y - start_v + n_warps) * sizeof(double);
  sum_of_squares_kernel<<<batch_size, 32 * n_warps, shared_mem_size, stream>>>(
    d_y, Tparams.mu, Tparams.ar, Tparams.ma, Tparams.sar, Tparams.sma,
    d_loglike, n_obs, n_phi, n_theta, order.p, order.q, order.P, order.Q,
    order.s, order.k, start_sum, start_y, start_v);
  CUDA_CHECK(hipPeekAtLastError());

  ML::POP_RANGE();
}

void batched_loglike(raft::handle_t& handle, const double* d_y, int batch_size,
                     int n_obs, const ARIMAOrder& order,
                     const ARIMAParams<double>& params, double* loglike,
                     double* d_vs, bool trans, bool host_loglike,
                     LoglikeMethod method, int truncate, int fc_steps,
                     double* d_fc, double level, double* d_lower,
                     double* d_upper) {
  ML::PUSH_RANGE(__func__);

  auto allocator = handle.get_device_allocator();
  auto stream = handle.get_stream();
  ARIMAParams<double> Tparams;

  ASSERT(method == MLE || fc_steps == 0,
         "Only MLE method is valid for forecasting");

  /* Create log-likelihood device array if host pointer is provided */
  double* d_loglike;
  MLCommon::device_buffer<double> loglike_buffer(allocator, stream);
  if (host_loglike) {
    loglike_buffer.resize(batch_size, stream);
    d_loglike = loglike_buffer.data();
  } else {
    d_loglike = loglike;
  }

  if (trans) {
    Tparams.allocate(order, batch_size, allocator, stream, true);

    MLCommon::TimeSeries::batched_jones_transform(
      order, batch_size, false, params, Tparams, allocator, stream);

    Tparams.mu = params.mu;
  } else {
    // non-transformed case: just use original parameters
    Tparams = params;
  }

  if (method == CSS) {
    conditional_sum_of_squares(handle, d_y, batch_size, n_obs, order, Tparams,
                               d_loglike, truncate);
  } else {
    batched_kalman_filter(handle, d_y, n_obs, Tparams, order, batch_size,
                          d_loglike, d_vs, fc_steps, d_fc, level, d_lower,
                          d_upper);
  }

  if (host_loglike) {
    /* Tranfer log-likelihood device -> host */
    raft::update_host(loglike, d_loglike, batch_size, stream);
  }

  if (trans) {
    Tparams.deallocate(order, batch_size, allocator, stream, true);
  }
  ML::POP_RANGE();
}

void batched_loglike(raft::handle_t& handle, const double* d_y, int batch_size,
                     int n_obs, const ARIMAOrder& order, const double* d_params,
                     double* loglike, double* d_vs, bool trans,
                     bool host_loglike, LoglikeMethod method, int truncate,
                     int fc_steps, double* d_fc, double level, double* d_lower,
                     double* d_upper) {
  ML::PUSH_RANGE(__func__);

  // unpack parameters
  auto allocator = handle.get_device_allocator();
  auto stream = handle.get_stream();
  ARIMAParams<double> params;
  params.allocate(order, batch_size, allocator, stream, false);
  params.unpack(order, batch_size, d_params, stream);

  batched_loglike(handle, d_y, batch_size, n_obs, order, params, loglike, d_vs,
                  trans, host_loglike, method, truncate, fc_steps, d_fc, level,
                  d_lower, d_upper);

  params.deallocate(order, batch_size, allocator, stream, false);

  ML::POP_RANGE();
}

void batched_loglike_grad(raft::handle_t& handle, const double* d_y,
                          int batch_size, int n_obs, const ARIMAOrder& order,
                          const double* d_x, double* d_grad, double h,
                          bool trans, LoglikeMethod method, int truncate) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.get_device_allocator();
  auto stream = handle.get_stream();
  auto counting = thrust::make_counting_iterator(0);
  int N = order.complexity();

  // Initialize the perturbed x vector
  MLCommon::device_buffer<double> x_pert(allocator, stream, N * batch_size);
  double* d_x_pert = x_pert.data();
  raft::copy(d_x_pert, d_x, N * batch_size, stream);

  // Create buffers for the log-likelihood and residuals
  MLCommon::device_buffer<double> ll_base(allocator, stream, batch_size);
  MLCommon::device_buffer<double> ll_pert(allocator, stream, batch_size);
  MLCommon::device_buffer<double> res(allocator, stream, n_obs * batch_size);
  double* d_ll_base = ll_base.data();
  double* d_ll_pert = ll_pert.data();

  // Evaluate the log-likelihood with the given parameter vector
  batched_loglike(handle, d_y, batch_size, n_obs, order, d_x, d_ll_base,
                  res.data(), trans, false, method, truncate);

  for (int i = 0; i < N; i++) {
    // Add the perturbation to the i-th parameter
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_x_pert[N * bid + i] = d_x[N * bid + i] + h;
                     });

    // Evaluate the log-likelihood with the positive perturbation
    batched_loglike(handle, d_y, batch_size, n_obs, order, d_x_pert, d_ll_pert,
                    res.data(), trans, false, method, truncate);

    // First derivative with a first-order accuracy
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       d_grad[N * bid + i] =
                         (d_ll_pert[bid] - d_ll_base[bid]) / h;
                     });

    // Reset the i-th parameter
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size,
      [=] __device__(int bid) { d_x_pert[N * bid + i] = d_x[N * bid + i]; });
  }

  ML::POP_RANGE();
}

void information_criterion(raft::handle_t& handle, const double* d_y,
                           int batch_size, int n_obs, const ARIMAOrder& order,
                           const ARIMAParams<double>& params, double* d_ic,
                           int ic_type) {
  ML::PUSH_RANGE(__func__);
  auto allocator = handle.get_device_allocator();
  auto stream = handle.get_stream();

  MLCommon::device_buffer<double> v_buffer(allocator, stream,
                                           n_obs * batch_size);

  /* Compute log-likelihood in d_ic */
  batched_loglike(handle, d_y, batch_size, n_obs, order, params, d_ic,
                  v_buffer.data(), false, false, MLE);

  /* Compute information criterion from log-likelihood and base term */
  MLCommon::Metrics::Batched::information_criterion(
    d_ic, d_ic, static_cast<MLCommon::Metrics::IC_Type>(ic_type),
    order.complexity(), batch_size, n_obs - order.n_diff(), stream);

  ML::POP_RANGE();
}

/**
 * Test that the parameters are valid for the inverse transform
 * 
 * @tparam isAr        Are these (S)AR or (S)MA parameters?
 * @param[in]  params  Parameters
 * @param[in]  pq      p for AR, q for MA, P for SAR, Q for SMA
 */
template <bool isAr>
DI bool test_invparams(const double* params, int pq) {
  double new_params[4];
  double tmp[4];

  constexpr double coef = isAr ? 1 : -1;

  for (int i = 0; i < pq; i++) {
    tmp[i] = params[i];
    new_params[i] = tmp[i];
  }

  // Perform inverse transform and stop before atanh step
  for (int j = pq - 1; j > 0; --j) {
    double a = new_params[j];
    for (int k = 0; k < j; ++k) {
      tmp[k] =
        (new_params[k] + coef * a * new_params[j - k - 1]) / (1 - (a * a));
    }
    for (int iter = 0; iter < j; ++iter) {
      new_params[iter] = tmp[iter];
    }
  }

  // Verify that the values are between -1 and 1
  bool result = true;
  for (int i = 0; i < pq; i++) {
    result = result && !(new_params[i] <= -1 || new_params[i] >= 1);
  }
  return result;
}

/**
 * Auxiliary function of _start_params: least square approximation of an
 * ARMA model (with or without seasonality)
 * @note: in this function the non-seasonal case has s=1, not s=0!
 */
void _arma_least_squares(raft::handle_t& handle, double* d_ar, double* d_ma,
                         double* d_sigma2,
                         const MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                         int p, int q, int s, bool estimate_sigma2, int k = 0,
                         double* d_mu = nullptr) {
  const auto& handle_impl = handle;
  auto stream = handle_impl.get_stream();
  auto cublas_handle = handle_impl.get_cublas_handle();
  auto allocator = handle_impl.get_device_allocator();
  auto counting = thrust::make_counting_iterator(0);

  int batch_size = bm_y.batches();
  int n_obs = bm_y.shape().first;

  int ps = p * s, qs = q * s;
  int p_ar = std::max(ps, 2 * qs);
  int r = std::max(p_ar + qs, ps);

  if ((q && p_ar >= n_obs - p_ar) || p + q + k >= n_obs - r) {
    // Too few observations for the estimate, fill with 0 (1 for sigma2)
    if (k)
      CUDA_CHECK(hipMemsetAsync(d_mu, 0, sizeof(double) * batch_size, stream));
    if (p)
      CUDA_CHECK(
        hipMemsetAsync(d_ar, 0, sizeof(double) * p * batch_size, stream));
    if (q)
      CUDA_CHECK(
        hipMemsetAsync(d_ma, 0, sizeof(double) * q * batch_size, stream));
    if (estimate_sigma2) {
      thrust::device_ptr<double> sigma2_thrust =
        thrust::device_pointer_cast(d_sigma2);
      thrust::fill(thrust::cuda::par.on(stream), sigma2_thrust,
                   sigma2_thrust + batch_size, 1.0);
    }
    return;
  }

  /* Matrix formed by lag matrices of y and the residuals respectively,
   * side by side. The left side will be used to estimate AR, the right
   * side to estimate MA */
  MLCommon::LinAlg::Batched::Matrix<double> bm_ls_ar_res(
    n_obs - r, p + q + k, batch_size, cublas_handle, allocator, stream, false);
  int ar_offset = r - ps;
  int res_offset = r - p_ar - qs;

  // Get residuals from an AR(p_ar) model to estimate the MA parameters
  if (q) {
    // Create lagged y
    int ls_height = n_obs - p_ar;
    MLCommon::LinAlg::Batched::Matrix<double> bm_ls =
      MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, p_ar);

    /* Matrix for the initial AR fit, initialized by copy of y
     * (note: this is because gels works in-place ; the matrix has larger
     *  dimensions than the actual AR fit) */
    MLCommon::LinAlg::Batched::Matrix<double> bm_ar_fit =
      MLCommon::LinAlg::Batched::b_2dcopy(bm_y, p_ar, 0, ls_height, 1);

    // Residual, initialized as offset y to avoid one kernel call
    MLCommon::LinAlg::Batched::Matrix<double> bm_residual(bm_ar_fit);

    // Initial AR fit
    MLCommon::LinAlg::Batched::b_gels(bm_ls, bm_ar_fit);

    // Compute residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, ls_height, 1, p_ar, -1.0,
                                      bm_ls, bm_ar_fit, 1.0, bm_residual);

    // Lags of the residual
    MLCommon::LinAlg::Batched::b_lagged_mat(bm_residual, bm_ls_ar_res, q,
                                            n_obs - r, res_offset,
                                            (n_obs - r) * (k + p), s);
  }

  // Fill the first column of the matrix with 1 if we fit an intercept
  if (k) {
    double* d_ls_ar_res = bm_ls_ar_res.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double* b_ls_ar_res =
                         d_ls_ar_res + bid * (n_obs - r) * (p + q + k);
                       for (int i = 0; i < n_obs - r; i++) {
                         b_ls_ar_res[i] = 1.0;
                       }
                     });
  }

  // Lags of y
  MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, bm_ls_ar_res, p, n_obs - r,
                                          ar_offset, (n_obs - r) * k, s);

  /* Initializing the vector for the ARMA fit
   * (note: also in-place as described for AR fit) */
  MLCommon::LinAlg::Batched::Matrix<double> bm_arma_fit =
    MLCommon::LinAlg::Batched::b_2dcopy(bm_y, r, 0, n_obs - r, 1);

  // The residuals will be computed only if sigma2 is requested
  MLCommon::LinAlg::Batched::Matrix<double> bm_final_residual(
    n_obs - r, 1, batch_size, cublas_handle, allocator, stream, false);
  if (estimate_sigma2) {
    raft::copy(bm_final_residual.raw_data(), bm_arma_fit.raw_data(),
               (n_obs - r) * batch_size, stream);
  }

  // ARMA fit
  MLCommon::LinAlg::Batched::b_gels(bm_ls_ar_res, bm_arma_fit);

  // Copy the results in the parameter vectors
  const double* d_arma_fit = bm_arma_fit.raw_data();
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     const double* b_arma_fit = d_arma_fit + bid * (n_obs - r);
                     if (k) {
                       d_mu[bid] = b_arma_fit[0];
                     }
                     if (p) {
                       double* b_ar = d_ar + bid * p;
                       for (int i = 0; i < p; i++) {
                         b_ar[i] = b_arma_fit[i + k];
                       }
                     }
                     if (q) {
                       double* b_ma = d_ma + bid * q;
                       for (int i = 0; i < q; i++) {
                         b_ma[i] = b_arma_fit[i + p + k];
                       }
                     }
                   });

  if (estimate_sigma2) {
    // Compute final residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false, false, n_obs - r, 1, p + q + k,
                                      -1.0, bm_ls_ar_res, bm_arma_fit, 1.0,
                                      bm_final_residual);

    // Compute variance
    double* d_residual = bm_final_residual.raw_data();
    thrust::for_each(thrust::cuda::par.on(stream), counting,
                     counting + batch_size, [=] __device__(int bid) {
                       double acc = 0.0;
                       const double* b_residual =
                         d_residual + (n_obs - r) * bid;
                       for (int i = q; i < n_obs - r; i++) {
                         double res = b_residual[i];
                         acc += res * res;
                       }
                       d_sigma2[bid] = acc / static_cast<double>(n_obs - r - q);
                     });
  }

  // If (S)AR or (S)MA are not valid for the inverse transform, set them to zero
  thrust::for_each(thrust::cuda::par.on(stream), counting,
                   counting + batch_size, [=] __device__(int bid) {
                     if (p) {
                       double* b_ar = d_ar + bid * p;
                       bool valid = test_invparams<true>(b_ar, p);
                       if (!valid) {
                         for (int ip = 0; ip < p; ip++) b_ar[ip] = 0;
                       }
                     }
                     if (q) {
                       double* b_ma = d_ma + bid * q;
                       bool valid = test_invparams<false>(b_ma, q);
                       if (!valid) {
                         for (int iq = 0; iq < q; iq++) b_ma[iq] = 0;
                       }
                     }
                   });
}

/**
 * Auxiliary function of estimate_x0: compute the starting parameters for
 * the series pre-processed by estimate_x0
 */
void _start_params(raft::handle_t& handle, ARIMAParams<double>& params,
                   const MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                   const ARIMAOrder& order) {
  // Estimate an ARMA fit without seasonality
  if (order.p + order.q + order.k)
    _arma_least_squares(handle, params.ar, params.ma, params.sigma2, bm_y,
                        order.p, order.q, 1, true, order.k, params.mu);

  // Estimate a seasonal ARMA fit independantly
  if (order.P + order.Q)
    _arma_least_squares(handle, params.sar, params.sma, params.sigma2, bm_y,
                        order.P, order.Q, order.s,
                        order.p + order.q + order.k == 0);
}

void estimate_x0(raft::handle_t& handle, ARIMAParams<double>& params,
                 const double* d_y, int batch_size, int n_obs,
                 const ARIMAOrder& order) {
  ML::PUSH_RANGE(__func__);
  const auto& handle_impl = handle;
  auto stream = handle_impl.get_stream();
  auto cublas_handle = handle_impl.get_cublas_handle();
  auto allocator = handle_impl.get_device_allocator();

  // Difference if necessary, copy otherwise
  MLCommon::LinAlg::Batched::Matrix<double> bm_yd(
    n_obs - order.d - order.s * order.D, 1, batch_size, cublas_handle,
    allocator, stream, false);
  MLCommon::TimeSeries::prepare_data(bm_yd.raw_data(), d_y, batch_size, n_obs,
                                     order.d, order.D, order.s, stream);

  // Do the computation of the initial parameters
  _start_params(handle, params, bm_yd, order);
  ML::POP_RANGE();
}

}  // namespace ML
