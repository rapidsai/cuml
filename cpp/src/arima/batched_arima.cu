#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/logical.h>

#include <cuml/tsa/batched_arima.hpp>
#include <cuml/tsa/batched_kalman.hpp>

#include <common/nvtx.hpp>
#include <linalg/batched/matrix.cuh>
#include <metrics/batched/information_criterion.cuh>
#include <raft/common/nvtx.hpp>
#include <raft/cuda_utils.cuh>
#include <raft/cudart_utils.h>
#include <raft/handle.hpp>
#include <raft/linalg/matrix_vector_op.hpp>
#include <rmm/device_uvector.hpp>
#include <timeSeries/arima_helpers.cuh>
#include <timeSeries/fillna.cuh>

namespace ML {

void pack(raft::handle_t& handle,
          const ARIMAParams<double>& params,
          const ARIMAOrder& order,
          int batch_size,
          double* param_vec)
{
  const auto stream = handle.get_stream();
  params.pack(order, batch_size, param_vec, stream);
}

void unpack(raft::handle_t& handle,
            ARIMAParams<double>& params,
            const ARIMAOrder& order,
            int batch_size,
            const double* param_vec)
{
  const auto stream = handle.get_stream();
  params.unpack(order, batch_size, param_vec, stream);
}

void batched_diff(raft::handle_t& handle,
                  double* d_y_diff,
                  const double* d_y,
                  int batch_size,
                  int n_obs,
                  const ARIMAOrder& order)
{
  const auto stream = handle.get_stream();
  MLCommon::TimeSeries::prepare_data(
    d_y_diff, d_y, batch_size, n_obs, order.d, order.D, order.s, stream);
}

template <typename T>
struct is_missing {
  typedef T argument_type;
  typedef T result_type;

  __thrust_exec_check_disable__ __device__ const T operator()(const T& x) const { return isnan(x); }
};  // end is_missing

bool detect_missing(raft::handle_t& handle, const double* d_y, int n_elem)
{
  return thrust::any_of(
    thrust::cuda::par.on(handle.get_stream()), d_y, d_y + n_elem, is_missing<double>());
}

void predict(raft::handle_t& handle,
             const ARIMAMemory<double>& arima_mem,
             const double* d_y,
             const double* d_exog,
             const double* d_exog_fut,
             int batch_size,
             int n_obs,
             int start,
             int end,
             const ARIMAOrder& order,
             const ARIMAParams<double>& params,
             double* d_y_p,
             bool pre_diff,
             double level,
             double* d_lower,
             double* d_upper)
{
  raft::common::nvtx::range fun_scope(__func__);
  const auto stream = handle.get_stream();

  bool diff     = order.need_diff() && pre_diff && level == 0;
  int num_steps = std::max(end - n_obs, 0);

  // Prepare data
  int n_obs_kf;
  const double* d_y_kf;
  const double* d_exog_kf;
  const double* d_exog_fut_kf = d_exog_fut;
  ARIMAOrder order_after_prep = order;
  rmm::device_uvector<double> exog_fut_buffer(0, stream);
  if (diff) {
    n_obs_kf = n_obs - order.n_diff();
    MLCommon::TimeSeries::prepare_data(
      arima_mem.y_diff, d_y, batch_size, n_obs, order.d, order.D, order.s, stream);
    if (order.n_exog > 0) {
      MLCommon::TimeSeries::prepare_data(arima_mem.exog_diff,
                                         d_exog,
                                         order.n_exog * batch_size,
                                         n_obs,
                                         order.d,
                                         order.D,
                                         order.s,
                                         stream);

      if (num_steps > 0) {
        exog_fut_buffer.resize(num_steps * order.n_exog * batch_size, stream);

        MLCommon::TimeSeries::prepare_future_data(exog_fut_buffer.data(),
                                                  d_exog,
                                                  d_exog_fut,
                                                  order.n_exog * batch_size,
                                                  n_obs,
                                                  num_steps,
                                                  order.d,
                                                  order.D,
                                                  order.s,
                                                  stream);

        d_exog_fut_kf = exog_fut_buffer.data();
      }
    }
    order_after_prep.d = 0;
    order_after_prep.D = 0;

    d_y_kf    = arima_mem.y_diff;
    d_exog_kf = arima_mem.exog_diff;
  } else {
    n_obs_kf  = n_obs;
    d_y_kf    = d_y;
    d_exog_kf = d_exog;
  }

  double* d_pred = arima_mem.pred;

  // Create temporary array for the forecasts
  rmm::device_uvector<double> fc_buffer(num_steps * batch_size, stream);
  double* d_y_fc = fc_buffer.data();

  // Compute the residual and forecast
  std::vector<double> loglike = std::vector<double>(batch_size);
  /// TODO: use device loglike to avoid useless copy ; part of #2233
  batched_loglike(handle,
                  arima_mem,
                  d_y_kf,
                  d_exog_kf,
                  batch_size,
                  n_obs_kf,
                  order_after_prep,
                  params,
                  loglike.data(),
                  false,
                  true,
                  MLE,
                  0,
                  num_steps,
                  d_y_fc,
                  d_exog_fut_kf,
                  level,
                  d_lower,
                  d_upper);

  auto counting  = thrust::make_counting_iterator(0);
  int predict_ld = end - start;

  //
  // In-sample prediction
  //

  // The prediction loop starts by filling undefined predictions with NaN,
  // then computes the predictions from the observations and residuals
  if (start < n_obs) {
    int res_offset = diff ? order.d + order.s * order.D : 0;
    int p_start    = std::max(start, res_offset);
    int p_end      = std::min(n_obs, end);
    int dD         = diff ? order.d + order.D : 0;
    int period1    = order.d ? 1 : order.s;
    int period2    = order.d == 2 ? 1 : order.s;

    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
        d_y_p[0] = 0.0;
        for (int i = 0; i < res_offset - start; i++) {
          d_y_p[bid * predict_ld + i] = nan("");
        }
        for (int i = p_start; i < p_end; i++) {
          if (dD == 0) {
            d_y_p[bid * predict_ld + i - start] = d_pred[bid * n_obs + i];
          } else if (dD == 1) {
            d_y_p[bid * predict_ld + i - start] =
              d_y[bid * n_obs + i - period1] + d_pred[bid * n_obs_kf + i - res_offset];
          } else {
            d_y_p[bid * predict_ld + i - start] =
              d_y[bid * n_obs + i - period1] + d_y[bid * n_obs + i - period2] -
              d_y[bid * n_obs + i - period1 - period2] + d_pred[bid * n_obs_kf + i - res_offset];
          }
        }
      });
  }

  //
  // Finalize out-of-sample forecast and copy in-sample predictions
  //

  if (num_steps) {
    if (diff) {
      MLCommon::TimeSeries::finalize_forecast(
        d_y_fc, d_y, num_steps, batch_size, n_obs, n_obs, order.d, order.D, order.s, stream);
    }

    // Copy forecast in d_y_p
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
        for (int i = 0; i < num_steps; i++) {
          d_y_p[bid * predict_ld + n_obs - start + i] = d_y_fc[num_steps * bid + i];
        }
      });
    /// TODO: 2D copy kernel?
  }
}

/**
 * Kernel to compute the sum-of-squares log-likelihood estimation
 *
 * @param[in]  d_y        Series to fit
 * @param[in]  d_mu       mu parameters
 * @param[in]  d_ar       AR parameters
 * @param[in]  d_ma       MA parameters
 * @param[in]  d_sar      Seasonal AR parameters
 * @param[in]  d_sma      Seasonal MA parameters
 * @param[out] d_loglike  Evaluated log-likelihood
 * @param[in]  n_obs      Number of observations in a time series
 * @param[in]  n_phi      Number of phi coefficients (combined AR-SAR)
 * @param[in]  n_theta    Number of theta coefficients (combined MA-SMA)
 * @param[in]  p          Number of AR parameters
 * @param[in]  q          Number of MA parameters
 * @param[in]  P          Number of seasonal AR parameters
 * @param[in]  Q          Number of seasonal MA parameters
 * @param[in]  s          Seasonal period or 0
 * @param[in]  k          Whether to use an intercept
 * @param[in]  start_sum  At which index to start the sum
 * @param[in]  start_y    First used y index (observation)
 * @param[in]  start_v    First used v index (residual)
 */
template <typename DataT>
__global__ void sum_of_squares_kernel(const DataT* d_y,
                                      const DataT* d_mu,
                                      const DataT* d_ar,
                                      const DataT* d_ma,
                                      const DataT* d_sar,
                                      const DataT* d_sma,
                                      DataT* d_loglike,
                                      int n_obs,
                                      int n_phi,
                                      int n_theta,
                                      int p,
                                      int q,
                                      int P,
                                      int Q,
                                      int s,
                                      int k,
                                      int start_sum,
                                      int start_y,
                                      int start_v)
{
  // Load phi, theta and mu to registers
  DataT phi, theta;
  if (threadIdx.x < n_phi) {
    phi = MLCommon::TimeSeries::reduced_polynomial<true>(
      blockIdx.x, d_ar, p, d_sar, P, s, threadIdx.x + 1);
  }
  if (threadIdx.x < n_theta) {
    theta = MLCommon::TimeSeries::reduced_polynomial<false>(
      blockIdx.x, d_ma, q, d_sma, Q, s, threadIdx.x + 1);
  }
  DataT mu = k ? d_mu[blockIdx.x] : (DataT)0;

  // Shared memory: load y and initialize the residuals
  extern __shared__ DataT shared_mem[];
  DataT* b_y  = shared_mem;
  DataT* b_vs = shared_mem + n_obs - start_y;
  for (int i = threadIdx.x; i < n_obs - start_y; i += blockDim.x) {
    b_y[i] = d_y[n_obs * blockIdx.x + i + start_y];
  }
  for (int i = threadIdx.x; i < start_sum - start_v; i += blockDim.x) {
    b_vs[i] = (DataT)0;
  }

  // Main loop
  char* temp_smem = (char*)(shared_mem + 2 * n_obs - start_y - start_v);
  DataT res, ssq = 0;
  for (int i = start_sum; i < n_obs; i++) {
    __syncthreads();
    res = (DataT)0;
    res -= threadIdx.x < n_phi ? phi * b_y[i - threadIdx.x - 1 - start_y] : (DataT)0;
    res -= threadIdx.x < n_theta ? theta * b_vs[i - threadIdx.x - 1 - start_v] : (DataT)0;
    res = raft::blockReduce(res, temp_smem);
    if (threadIdx.x == 0) {
      res += b_y[i - start_y] - mu;
      b_vs[i - start_v] = res;
      ssq += res * res;
    }
  }

  // Compute log-likelihood and write it to global memory
  if (threadIdx.x == 0) {
    d_loglike[blockIdx.x] =
      -0.5 * static_cast<DataT>(n_obs) * raft::myLog(ssq / static_cast<DataT>(n_obs - start_sum));
  }
}

/**
 * Sum-of-squares estimation method
 *
 * @param[in]  handle     cuML handle
 * @param[in]  d_y        Series to fit: shape = (n_obs, batch_size)
 * @param[in]  batch_size Number of time series
 * @param[in]  n_obs      Number of observations in a time series
 * @param[in]  order      ARIMA hyper-parameters
 * @param[in]  Tparams    Transformed parameters
 * @param[out] d_loglike  Evaluated log-likelihood (device)
 * @param[in]  truncate   Number of observations to skip in the sum
 */
void conditional_sum_of_squares(raft::handle_t& handle,
                                const double* d_y,
                                int batch_size,
                                int n_obs,
                                const ARIMAOrder& order,
                                const ARIMAParams<double>& Tparams,
                                double* d_loglike,
                                int truncate)
{
  raft::common::nvtx::range fun_scope(__func__);
  auto stream = handle.get_stream();

  int n_phi     = order.n_phi();
  int n_theta   = order.n_theta();
  int max_lags  = std::max(n_phi, n_theta);
  int start_sum = std::max(max_lags, truncate);
  int start_y   = start_sum - n_phi;
  int start_v   = start_sum - n_theta;

  // Compute the sum-of-squares and the log-likelihood
  int n_warps            = std::max(raft::ceildiv<int>(max_lags, 32), 1);
  size_t shared_mem_size = (2 * n_obs - start_y - start_v + n_warps) * sizeof(double);
  sum_of_squares_kernel<<<batch_size, 32 * n_warps, shared_mem_size, stream>>>(d_y,
                                                                               Tparams.mu,
                                                                               Tparams.ar,
                                                                               Tparams.ma,
                                                                               Tparams.sar,
                                                                               Tparams.sma,
                                                                               d_loglike,
                                                                               n_obs,
                                                                               n_phi,
                                                                               n_theta,
                                                                               order.p,
                                                                               order.q,
                                                                               order.P,
                                                                               order.Q,
                                                                               order.s,
                                                                               order.k,
                                                                               start_sum,
                                                                               start_y,
                                                                               start_v);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

void batched_loglike(raft::handle_t& handle,
                     const ARIMAMemory<double>& arima_mem,
                     const double* d_y,
                     const double* d_exog,
                     int batch_size,
                     int n_obs,
                     const ARIMAOrder& order,
                     const ARIMAParams<double>& params,
                     double* loglike,
                     bool trans,
                     bool host_loglike,
                     LoglikeMethod method,
                     int truncate,
                     int fc_steps,
                     double* d_fc,
                     const double* d_exog_fut,
                     double level,
                     double* d_lower,
                     double* d_upper)
{
  raft::common::nvtx::range fun_scope(__func__);

  auto stream = handle.get_stream();

  double* d_pred = arima_mem.pred;

  ARIMAParams<double> Tparams = {params.mu,
                                 params.beta,
                                 arima_mem.Tparams_ar,
                                 arima_mem.Tparams_ma,
                                 arima_mem.Tparams_sar,
                                 arima_mem.Tparams_sma,
                                 arima_mem.Tparams_sigma2};

  ASSERT(method == MLE || fc_steps == 0, "Only MLE method is valid for forecasting");

  /* Create log-likelihood device array if host pointer is provided */
  double* d_loglike = host_loglike ? arima_mem.loglike : loglike;

  if (trans) {
    MLCommon::TimeSeries::batched_jones_transform(
      order, batch_size, false, params, Tparams, stream);
  } else {
    // non-transformed case: just use original parameters
    Tparams.ar     = params.ar;
    Tparams.ma     = params.ma;
    Tparams.sar    = params.sar;
    Tparams.sma    = params.sma;
    Tparams.sigma2 = params.sigma2;
  }

  if (method == CSS) {
    conditional_sum_of_squares(handle, d_y, batch_size, n_obs, order, Tparams, d_loglike, truncate);
  } else {
    batched_kalman_filter(handle,
                          arima_mem,
                          d_y,
                          d_exog,
                          n_obs,
                          Tparams,
                          order,
                          batch_size,
                          d_loglike,
                          d_pred,
                          fc_steps,
                          d_fc,
                          d_exog_fut,
                          level,
                          d_lower,
                          d_upper);
  }

  if (host_loglike) {
    /* Tranfer log-likelihood device -> host */
    raft::update_host(loglike, d_loglike, batch_size, stream);
  }
}

void batched_loglike(raft::handle_t& handle,
                     const ARIMAMemory<double>& arima_mem,
                     const double* d_y,
                     const double* d_exog,
                     int batch_size,
                     int n_obs,
                     const ARIMAOrder& order,
                     const double* d_params,
                     double* loglike,
                     bool trans,
                     bool host_loglike,
                     LoglikeMethod method,
                     int truncate)
{
  raft::common::nvtx::range fun_scope(__func__);

  // unpack parameters
  auto stream = handle.get_stream();

  ARIMAParams<double> params = {arima_mem.params_mu,
                                arima_mem.params_beta,
                                arima_mem.params_ar,
                                arima_mem.params_ma,
                                arima_mem.params_sar,
                                arima_mem.params_sma,
                                arima_mem.params_sigma2};

  params.unpack(order, batch_size, d_params, stream);

  batched_loglike(handle,
                  arima_mem,
                  d_y,
                  d_exog,
                  batch_size,
                  n_obs,
                  order,
                  params,
                  loglike,
                  trans,
                  host_loglike,
                  method,
                  truncate);
}

void batched_loglike_grad(raft::handle_t& handle,
                          const ARIMAMemory<double>& arima_mem,
                          const double* d_y,
                          const double* d_exog,
                          int batch_size,
                          int n_obs,
                          const ARIMAOrder& order,
                          const double* d_x,
                          double* d_grad,
                          double h,
                          bool trans,
                          LoglikeMethod method,
                          int truncate)
{
  raft::common::nvtx::range fun_scope(__func__);
  auto stream   = handle.get_stream();
  auto counting = thrust::make_counting_iterator(0);
  int N         = order.complexity();

  // Initialize the perturbed x vector
  double* d_x_pert = arima_mem.x_pert;
  raft::copy(d_x_pert, d_x, N * batch_size, stream);

  double* d_ll_base = arima_mem.loglike_base;
  double* d_ll_pert = arima_mem.loglike_pert;

  // Evaluate the log-likelihood with the given parameter vector
  batched_loglike(handle,
                  arima_mem,
                  d_y,
                  d_exog,
                  batch_size,
                  n_obs,
                  order,
                  d_x,
                  d_ll_base,
                  trans,
                  false,
                  method,
                  truncate);

  for (int i = 0; i < N; i++) {
    // Add the perturbation to the i-th parameter
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
        d_x_pert[N * bid + i] = d_x[N * bid + i] + h;
      });

    // Evaluate the log-likelihood with the positive perturbation
    batched_loglike(handle,
                    arima_mem,
                    d_y,
                    d_exog,
                    batch_size,
                    n_obs,
                    order,
                    d_x_pert,
                    d_ll_pert,
                    trans,
                    false,
                    method,
                    truncate);

    // First derivative with a first-order accuracy
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
        d_grad[N * bid + i] = (d_ll_pert[bid] - d_ll_base[bid]) / h;
      });

    // Reset the i-th parameter
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
        d_x_pert[N * bid + i] = d_x[N * bid + i];
      });
  }
}

void information_criterion(raft::handle_t& handle,
                           const ARIMAMemory<double>& arima_mem,
                           const double* d_y,
                           const double* d_exog,
                           int batch_size,
                           int n_obs,
                           const ARIMAOrder& order,
                           const ARIMAParams<double>& params,
                           double* d_ic,
                           int ic_type)
{
  raft::common::nvtx::range fun_scope(__func__);
  auto stream = handle.get_stream();

  /* Compute log-likelihood in d_ic */
  batched_loglike(
    handle, arima_mem, d_y, d_exog, batch_size, n_obs, order, params, d_ic, false, false, MLE);

  /* Compute information criterion from log-likelihood and base term */
  MLCommon::Metrics::Batched::information_criterion(
    d_ic,
    d_ic,
    static_cast<MLCommon::Metrics::IC_Type>(ic_type),
    order.complexity(),
    batch_size,
    n_obs - order.n_diff(),
    stream);
}

/**
 * Test that the parameters are valid for the inverse transform
 *
 * @tparam isAr        Are these (S)AR or (S)MA parameters?
 * @param[in]  params  Parameters
 * @param[in]  pq      p for AR, q for MA, P for SAR, Q for SMA
 */
template <bool isAr>
DI bool test_invparams(const double* params, int pq)
{
  double new_params[8];
  double tmp[8];

  constexpr double coef = isAr ? 1 : -1;

  for (int i = 0; i < pq; i++) {
    tmp[i]        = params[i];
    new_params[i] = tmp[i];
  }

  // Perform inverse transform and stop before atanh step
  for (int j = pq - 1; j > 0; --j) {
    double a = new_params[j];
    for (int k = 0; k < j; ++k) {
      tmp[k] = (new_params[k] + coef * a * new_params[j - k - 1]) / (1 - (a * a));
    }
    for (int iter = 0; iter < j; ++iter) {
      new_params[iter] = tmp[iter];
    }
  }

  // Verify that the values are between -1 and 1
  bool result = true;
  for (int i = 0; i < pq; i++) {
    result = result && !(new_params[i] <= -1 || new_params[i] >= 1);
  }
  return result;
}

/**
 * Auxiliary function of _start_params: least square approximation of an
 * ARMA model (with or without seasonality)
 * @note: in this function the non-seasonal case has s=1, not s=0!
 */
void _arma_least_squares(raft::handle_t& handle,
                         double* d_ar,
                         double* d_ma,
                         double* d_sigma2,
                         const MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                         int p,
                         int q,
                         int s,
                         bool estimate_sigma2,
                         int k        = 0,
                         double* d_mu = nullptr)
{
  const auto& handle_impl = handle;
  auto stream             = handle_impl.get_stream();
  auto cublas_handle      = handle_impl.get_cublas_handle();
  auto counting           = thrust::make_counting_iterator(0);

  int batch_size = bm_y.batches();
  int n_obs      = bm_y.shape().first;

  int ps = p * s, qs = q * s;
  int p_ar = std::max(ps, 2 * qs);
  int r    = std::max(p_ar + qs, ps);

  if ((q && p_ar >= n_obs - p_ar) || p + q + k >= n_obs - r) {
    // Too few observations for the estimate, fill with 0 (1 for sigma2)
    if (k) RAFT_CUDA_TRY(hipMemsetAsync(d_mu, 0, sizeof(double) * batch_size, stream));
    if (p) RAFT_CUDA_TRY(hipMemsetAsync(d_ar, 0, sizeof(double) * p * batch_size, stream));
    if (q) RAFT_CUDA_TRY(hipMemsetAsync(d_ma, 0, sizeof(double) * q * batch_size, stream));
    if (estimate_sigma2) {
      thrust::device_ptr<double> sigma2_thrust = thrust::device_pointer_cast(d_sigma2);
      thrust::fill(thrust::cuda::par.on(stream), sigma2_thrust, sigma2_thrust + batch_size, 1.0);
    }
    return;
  }

  /* Matrix formed by lag matrices of y and the residuals respectively,
   * side by side. The left side will be used to estimate AR, the right
   * side to estimate MA */
  MLCommon::LinAlg::Batched::Matrix<double> bm_ls_ar_res(
    n_obs - r, p + q + k, batch_size, cublas_handle, stream, false);
  int ar_offset  = r - ps;
  int res_offset = r - p_ar - qs;

  // Get residuals from an AR(p_ar) model to estimate the MA parameters
  if (q) {
    // Create lagged y
    int ls_height = n_obs - p_ar;
    MLCommon::LinAlg::Batched::Matrix<double> bm_ls =
      MLCommon::LinAlg::Batched::b_lagged_mat(bm_y, p_ar);

    /* Matrix for the initial AR fit, initialized by copy of y
     * (note: this is because gels works in-place ; the matrix has larger
     *  dimensions than the actual AR fit) */
    MLCommon::LinAlg::Batched::Matrix<double> bm_ar_fit =
      MLCommon::LinAlg::Batched::b_2dcopy(bm_y, p_ar, 0, ls_height, 1);

    // Residual, initialized as offset y to avoid one kernel call
    MLCommon::LinAlg::Batched::Matrix<double> bm_residual(bm_ar_fit);

    // Initial AR fit
    MLCommon::LinAlg::Batched::b_gels(bm_ls, bm_ar_fit);

    // Compute residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(
      false, false, ls_height, 1, p_ar, -1.0, bm_ls, bm_ar_fit, 1.0, bm_residual);

    // Lags of the residual
    MLCommon::LinAlg::Batched::b_lagged_mat(
      bm_residual, bm_ls_ar_res, q, n_obs - r, res_offset, (n_obs - r) * (k + p), s);
  }

  // Fill the first column of the matrix with 1 if we fit an intercept
  if (k) {
    double* d_ls_ar_res = bm_ls_ar_res.raw_data();
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
        double* b_ls_ar_res = d_ls_ar_res + bid * (n_obs - r) * (p + q + k);
        for (int i = 0; i < n_obs - r; i++) {
          b_ls_ar_res[i] = 1.0;
        }
      });
  }

  // Lags of y
  MLCommon::LinAlg::Batched::b_lagged_mat(
    bm_y, bm_ls_ar_res, p, n_obs - r, ar_offset, (n_obs - r) * k, s);

  /* Initializing the vector for the ARMA fit
   * (note: also in-place as described for AR fit) */
  MLCommon::LinAlg::Batched::Matrix<double> bm_arma_fit =
    MLCommon::LinAlg::Batched::b_2dcopy(bm_y, r, 0, n_obs - r, 1);

  // The residuals will be computed only if sigma2 is requested
  MLCommon::LinAlg::Batched::Matrix<double> bm_final_residual(
    n_obs - r, 1, batch_size, cublas_handle, stream, false);
  if (estimate_sigma2) {
    raft::copy(
      bm_final_residual.raw_data(), bm_arma_fit.raw_data(), (n_obs - r) * batch_size, stream);
  }

  // ARMA fit
  MLCommon::LinAlg::Batched::b_gels(bm_ls_ar_res, bm_arma_fit);

  // Copy the results in the parameter vectors
  const double* d_arma_fit = bm_arma_fit.raw_data();
  thrust::for_each(
    thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
      const double* b_arma_fit = d_arma_fit + bid * (n_obs - r);
      if (k) { d_mu[bid] = b_arma_fit[0]; }
      if (p) {
        double* b_ar = d_ar + bid * p;
        for (int i = 0; i < p; i++) {
          b_ar[i] = b_arma_fit[i + k];
        }
      }
      if (q) {
        double* b_ma = d_ma + bid * q;
        for (int i = 0; i < q; i++) {
          b_ma[i] = b_arma_fit[i + p + k];
        }
      }
    });

  if (estimate_sigma2) {
    // Compute final residual (technically a gemv)
    MLCommon::LinAlg::Batched::b_gemm(false,
                                      false,
                                      n_obs - r,
                                      1,
                                      p + q + k,
                                      -1.0,
                                      bm_ls_ar_res,
                                      bm_arma_fit,
                                      1.0,
                                      bm_final_residual);

    // Compute variance
    double* d_residual = bm_final_residual.raw_data();
    thrust::for_each(
      thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
        double acc               = 0.0;
        const double* b_residual = d_residual + (n_obs - r) * bid;
        for (int i = q; i < n_obs - r; i++) {
          double res = b_residual[i];
          acc += res * res;
        }
        d_sigma2[bid] = acc / static_cast<double>(n_obs - r - q);
      });
  }

  // If (S)AR or (S)MA are not valid for the inverse transform, set them to zero
  thrust::for_each(
    thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
      if (p) {
        double* b_ar = d_ar + bid * p;
        bool valid   = test_invparams<true>(b_ar, p);
        if (!valid) {
          for (int ip = 0; ip < p; ip++)
            b_ar[ip] = 0;
        }
      }
      if (q) {
        double* b_ma = d_ma + bid * q;
        bool valid   = test_invparams<false>(b_ma, q);
        if (!valid) {
          for (int iq = 0; iq < q; iq++)
            b_ma[iq] = 0;
        }
      }
    });
}

/**
 * Auxiliary function of estimate_x0: compute the starting parameters for
 * the series pre-processed by estimate_x0
 */
void _start_params(raft::handle_t& handle,
                   ARIMAParams<double>& params,
                   MLCommon::LinAlg::Batched::Matrix<double>& bm_y,
                   const MLCommon::LinAlg::Batched::Matrix<double>& bm_exog,
                   const ARIMAOrder& order)
{
  int batch_size      = bm_exog.batches();
  hipStream_t stream = bm_exog.stream();

  // Estimate exog coefficients and subtract component to endog.
  // Exog coefficients are estimated by fitting a linear regression with X=exog, y=endog
  if (order.n_exog > 0) {
    // In most cases, the system will be overdetermined and we can use gels
    if (bm_exog.shape().first > static_cast<unsigned int>(order.n_exog)) {
      // Make a copy of the exogenous series for in-place gels
      MLCommon::LinAlg::Batched::Matrix<double> bm_exog_copy(bm_exog);
      // Make a copy of the endogenous series for in-place gels
      MLCommon::LinAlg::Batched::Matrix<double> bm_y_copy(bm_y);

      // Least-squares solution of overdetermined system
      rmm::device_uvector<int> info(batch_size, stream);
      b_gels(bm_exog_copy, bm_y_copy, info.data());

      // Make a batched matrix around the exogenous coefficients
      rmm::device_uvector<double*> beta_pointers(batch_size, stream);
      MLCommon::LinAlg::Batched::Matrix<double> bm_exog_coef(order.n_exog,
                                                             1,
                                                             batch_size,
                                                             bm_exog.cublasHandle(),
                                                             beta_pointers.data(),
                                                             params.beta,
                                                             stream,
                                                             false);

      // Copy the solution of the system to the parameters array
      b_2dcopy(bm_y_copy, bm_exog_coef, 0, 0, order.n_exog, 1);

      // Set parameters to zero when solving was not successful
      auto counting       = thrust::make_counting_iterator(0);
      int* devInfoArray   = info.data();
      double* d_exog_coef = bm_exog_coef.raw_data();
      const int& n_exog   = order.n_exog;
      thrust::for_each(
        thrust::cuda::par.on(stream), counting, counting + batch_size, [=] __device__(int bid) {
          if (devInfoArray[bid] > 0) {
            for (int i = 0; i < n_exog; i++) {
              d_exog_coef[bid * n_exog + i] = 0.0;
            }
          }
        });

      // Compute exogenous component and store the result in bm_y_copy
      b_gemm(false,
             false,
             bm_exog.shape().first,
             1,
             bm_exog.shape().second,
             1.0,
             bm_exog,
             bm_exog_coef,
             0.0,
             bm_y_copy);

      // Subtract exogenous component to endogenous variable
      b_aA_op_B(bm_y, bm_y_copy, bm_y, [] __device__(double a, double b) { return a - b; });
    }
    // In other cases, we initialize to zero
    else {
      RAFT_CUDA_TRY(
        hipMemsetAsync(params.beta, 0, order.n_exog * batch_size * sizeof(double), stream));
    }
  }

  // Estimate an ARMA fit without seasonality
  if (order.p + order.q + order.k)
    _arma_least_squares(handle,
                        params.ar,
                        params.ma,
                        params.sigma2,
                        bm_y,
                        order.p,
                        order.q,
                        1,
                        true,
                        order.k,
                        params.mu);

  // Estimate a seasonal ARMA fit independantly
  if (order.P + order.Q)
    _arma_least_squares(handle,
                        params.sar,
                        params.sma,
                        params.sigma2,
                        bm_y,
                        order.P,
                        order.Q,
                        order.s,
                        order.p + order.q + order.k == 0);
}

void estimate_x0(raft::handle_t& handle,
                 ARIMAParams<double>& params,
                 const double* d_y,
                 const double* d_exog,
                 int batch_size,
                 int n_obs,
                 const ARIMAOrder& order,
                 bool missing)
{
  raft::common::nvtx::range fun_scope(__func__);
  const auto& handle_impl = handle;
  auto stream             = handle_impl.get_stream();
  auto cublas_handle      = handle_impl.get_cublas_handle();

  /// TODO: solve exogenous coefficients with only valid rows instead of interpolation?
  // Pros: better coefficients
  // Cons: harder to test, a bit more complicated

  // Least squares can't deal with missing values: create copy with naive
  // replacements for missing values
  const double* d_y_no_missing;
  rmm::device_uvector<double> y_no_missing(0, stream);
  if (missing) {
    y_no_missing.resize(n_obs * batch_size, stream);
    d_y_no_missing = y_no_missing.data();

    raft::copy(y_no_missing.data(), d_y, n_obs * batch_size, stream);
    MLCommon::TimeSeries::fillna(y_no_missing.data(), batch_size, n_obs, stream);
  } else {
    d_y_no_missing = d_y;
  }

  // Difference if necessary, copy otherwise
  MLCommon::LinAlg::Batched::Matrix<double> bm_yd(
    n_obs - order.d - order.s * order.D, 1, batch_size, cublas_handle, stream, false);
  MLCommon::TimeSeries::prepare_data(
    bm_yd.raw_data(), d_y_no_missing, batch_size, n_obs, order.d, order.D, order.s, stream);

  // Difference or copy exog
  MLCommon::LinAlg::Batched::Matrix<double> bm_exog_diff(
    n_obs - order.d - order.s * order.D, order.n_exog, batch_size, cublas_handle, stream, false);
  if (order.n_exog > 0) {
    MLCommon::TimeSeries::prepare_data(bm_exog_diff.raw_data(),
                                       d_exog,
                                       order.n_exog * batch_size,
                                       n_obs,
                                       order.d,
                                       order.D,
                                       order.s,
                                       stream);
  }

  // Do the computation of the initial parameters
  _start_params(handle, params, bm_yd, bm_exog_diff, order);
}

}  // namespace ML
