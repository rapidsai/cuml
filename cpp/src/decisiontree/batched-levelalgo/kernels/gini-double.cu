#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/binary_search.h>
#include <common/grid_sync.cuh>
#include <cstdio>
#include <hipcub/hipcub.hpp>
#include <raft/cuda_utils.cuh>
#include <cuml/tree/flatnode.h>
#include "builder_kernels.cuh"

namespace ML {
namespace DT {
template
__global__ void nodeSplitKernel<double, int, int, GiniObjectiveFunction<double, int, int>, TPB_DEFAULT>(int max_depth,
                                int min_samples_leaf,
                                int min_samples_split,
                                int max_leaves,
                                double min_impurity_decrease,
                                Input<double, int, int> input,
                                NodeWorkItem* work_items,
                                const Split<double, int>* splits);
template __global__ void leafKernel< Input<double, int, int>,  SparseTreeNode<double, int, int>,  GiniObjectiveFunction<double, int, int>,  double>
(GiniObjectiveFunction<double, int, int> objective,
                           Input<double, int, int> input,
                           const SparseTreeNode<double, int, int>* tree,
                           const InstanceRange* instance_ranges,
                           double* leaves);
template
__global__ void computeSplitKernel< double,
           int,
           int,
           TPB_DEFAULT,
           GiniObjectiveFunction<double, int, int>,
           CountBin>(CountBin* hist,
                                   int nbins,
                                   int max_depth,
                                   int min_samples_split,
                                   int max_leaves,
                                   Input<double, int, int> input,
                                   const NodeWorkItem* work_items,
                                   int colStart,
                                   int* done_count,
                                   int* mutex,
                                   volatile Split<double, int>* splits,
                                   GiniObjectiveFunction<double, int, int> objective,
                                   int treeid,
                                   const WorkloadInfo<int>* workload_info,
                                   uint64_t seed);
}  // namespace DT
}  // namespace ML
