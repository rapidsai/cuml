#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/binary_search.h>
#include <common/grid_sync.cuh>
#include <cstdio>
#include <hipcub/hipcub.hpp>
#include <raft/cuda_utils.cuh>
#include <cuml/tree/flatnode.h>
#include "kernels.cuh"

namespace ML {
namespace DT {
template
__global__ void nodeSplitKernel<float, int, int, GiniObjectiveFunction<float, int, int>, TPB_DEFAULT>(int max_depth,
                                int min_samples_leaf,
                                int min_samples_split,
                                int max_leaves,
                                float min_impurity_decrease,
                                Input<float, int, int> input,
                                NodeWorkItem* work_items,
                                const Split<float, int>* splits);
template __global__ void leafKernel< Input<float, int, int>,  SparseTreeNode<float, int, int>,  GiniObjectiveFunction<float, int, int>,  float>
(GiniObjectiveFunction<float, int, int> objective,
                           Input<float, int, int> input,
                           const SparseTreeNode<float, int, int>* tree,
                           const InstanceRange* instance_ranges,
                           float* leaves);
template
__global__ void computeSplitKernel< float,
           int,
           int,
           TPB_DEFAULT,
           GiniObjectiveFunction<float, int, int>,
           CountBin>(CountBin* hist,
                                   int nbins,
                                   int max_depth,
                                   int min_samples_split,
                                   int max_leaves,
                                   Input<float, int, int> input,
                                   const NodeWorkItem* work_items,
                                   int colStart,
                                   int* done_count,
                                   int* mutex,
                                   volatile Split<float, int>* splits,
                                   GiniObjectiveFunction<float, int, int> objective,
                                   int treeid,
                                   const WorkloadInfo<int>* workload_info,
                                   uint64_t seed);
}  // namespace DT
}  // namespace ML
