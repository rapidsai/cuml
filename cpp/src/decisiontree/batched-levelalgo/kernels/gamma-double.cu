/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/tree/flatnode.h>

#include "../bins.cuh"
#include "../objectives.cuh"

namespace ML {
namespace DT {
using _DataT      = double;
using _LabelT     = double;
using _IdxT       = int;
using _ObjectiveT = GammaObjectiveFunction<_DataT, _LabelT, _IdxT>;
using _BinT       = AggregateBin;
using _DatasetT   = Dataset<_DataT, _LabelT, _IdxT>;
using _NodeT      = SparseTreeNode<_DataT, _LabelT, _IdxT>;
}  // namespace DT
}  // namespace ML

#include "builder_kernels_impl.cuh"
