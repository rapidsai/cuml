#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/binary_search.h>
#include <common/grid_sync.cuh>
#include <cstdio>
#include <hipcub/hipcub.hpp>
#include <raft/cuda_utils.cuh>
#include <cuml/tree/flatnode.h>
#include "builder_kernels.cuh"

namespace ML {
namespace DT {
template
__global__ void nodeSplitKernel<double, double, int, InverseGaussianObjectiveFunction<double, double, int>, TPB_DEFAULT>(int max_depth,
                                int min_samples_leaf,
                                int min_samples_split,
                                int max_leaves,
                                double min_impurity_decrease,
                                Input<double, double, int> input,
                                NodeWorkItem* work_items,
                                const Split<double, int>* splits);
template __global__ void leafKernel< Input<double, double, int>,  SparseTreeNode<double, double, int>,  InverseGaussianObjectiveFunction<double, double, int>,  double>
(InverseGaussianObjectiveFunction<double, double, int> objective,
                           Input<double, double, int> input,
                           const SparseTreeNode<double, double, int>* tree,
                           const InstanceRange* instance_ranges,
                           double* leaves);
template
__global__ void computeSplitKernel< double,
           double,
           int,
           TPB_DEFAULT,
           InverseGaussianObjectiveFunction<double, double, int>,
           AggregateBin>(AggregateBin* hist,
                                   int nbins,
                                   int max_depth,
                                   int min_samples_split,
                                   int max_leaves,
                                   Input<double, double, int> input,
                                   const NodeWorkItem* work_items,
                                   int colStart,
                                   int* done_count,
                                   int* mutex,
                                   volatile Split<double, int>* splits,
                                   InverseGaussianObjectiveFunction<double, double, int> objective,
                                   int treeid,
                                   const WorkloadInfo<int>* workload_info,
                                   uint64_t seed);
}  // namespace DT
}  // namespace ML
