#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "../quantiles.cuh"

namespace ML {
namespace DT {

template <typename T>
__global__ void computeQuantilesSorted(T* quantiles,
                                       const int n_bins,
                                       const T* sorted_data,
                                       const int length)
{
  int tid          = threadIdx.x + blockIdx.x * blockDim.x;
  double bin_width = static_cast<double>(length) / n_bins;
  int index        = int(round((tid + 1) * bin_width)) - 1;
  index            = min(max(0, index), length - 1);
  if (tid < n_bins) { quantiles[tid] = sorted_data[index]; }

  return;
}

// instantiation
template __global__ void computeQuantilesSorted<float>(float* quantiles,
                                                       const int n_bins,
                                                       const float* sorted_data,
                                                       const int length);
template __global__ void computeQuantilesSorted<double>(double* quantiles,
                                                        const int n_bins,
                                                        const double* sorted_data,
                                                        const int length);

}  // end namespace DT
}  // end namespace ML