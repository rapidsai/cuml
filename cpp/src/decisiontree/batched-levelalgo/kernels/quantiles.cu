#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "../quantiles.cuh"

namespace ML {
namespace DT {

template <typename T>
__global__ void computeQuantilesKernel(
  T* quantiles, int* n_bins_unique, const T* sorted_data, const int n_bins_max, const int n_rows)
{
  // extern __shared__ char smem[];
  // auto* smem_quantiles = (T*)smem;
  // __shared__ int n_unique_bins;
  // int col          = blockIdx.x;  // each col per block
  // int base         = col * n_rows;
  double bin_width = static_cast<double>(n_rows) / n_bins_max;

  for (int bin = threadIdx.x; bin < n_bins_max; bin += blockDim.x) {
    // get index by interpolation
    int idx        = int(round((bin + 1) * bin_width)) - 1;
    idx            = min(max(0, idx), n_rows - 1);
    quantiles[bin] = sorted_data[idx];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    // make quantiles unique, in-place
    auto new_last = thrust::unique(thrust::device, quantiles, quantiles + n_bins_max);
    // get the unique count
    *n_bins_unique = new_last - quantiles;
  }

  __syncthreads();
  return;
}

// instantiation
template __global__ void computeQuantilesKernel<float>(float* quantiles,
                                                       int* n_bins_unique,
                                                       const float* sorted_data,
                                                       const int n_bins_max,
                                                       const int n_rows);
template __global__ void computeQuantilesKernel<double>(double* quantiles,
                                                        int* n_bins_unique,
                                                        const double* sorted_data,
                                                        const int n_bins_max,
                                                        const int n_rows);

}  // end namespace DT
}  // end namespace ML