#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "../quantiles.cuh"

namespace ML {
namespace DT {

template <typename T>
__global__ void computeQuantilesKernel(
  T* quantiles, int* n_bins, const T* sorted_data, const int max_n_bins, const int n_rows)
{
  double bin_width = static_cast<double>(n_rows) / max_n_bins;

  for (int bin = threadIdx.x; bin < max_n_bins; bin += blockDim.x) {
    // get index by interpolation
    int idx        = int(round((bin + 1) * bin_width)) - 1;
    idx            = min(max(0, idx), n_rows - 1);
    quantiles[bin] = sorted_data[idx];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    // make quantiles unique, in-place
    // thrust::seq to explicitly disable cuda dynamic parallelism here
    auto new_last = thrust::unique(thrust::seq, quantiles, quantiles + max_n_bins);
    // get the unique count
    *n_bins = new_last - quantiles;
  }

  __syncthreads();
  return;
}

// instantiation
template __global__ void computeQuantilesKernel<float>(
  float* quantiles, int* n_bins, const float* sorted_data, const int max_n_bins, const int n_rows);
template __global__ void computeQuantilesKernel<double>(double* quantiles,
                                                        int* n_bins,
                                                        const double* sorted_data,
                                                        const int max_n_bins,
                                                        const int n_rows);

}  // end namespace DT
}  // end namespace ML
