#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/common/device_buffer.hpp>
#include <cuml/tree/decisiontree.hpp>
#include <raft/handle.hpp>

#include <cuml/tree/flatnode.h>
#include "decisiontree_impl.cuh"

namespace ML {
namespace DecisionTree {

/**
 * @brief Set all DecisionTreeParams members.
 * @param[in,out] params: update with tree parameters
 * @param[in] cfg_max_depth: maximum tree depth; default -1
 * @param[in] cfg_max_leaves: maximum leaves; default -1
 * @param[in] cfg_max_features: maximum number of features; default 1.0f
 * @param[in] cfg_n_bins: number of bins; default 8
 * @param[in] cfg_min_samples_leaf: min. rows in each leaf node; default 1
 * @param[in] cfg_min_samples_split: min. rows needed to split an internal node;
 *            default 2
 * @param[in] cfg_split_criterion: split criterion; default CRITERION_END,
 *            i.e., GINI for classification or MSE for regression
 * @param[in] cfg_max_batch_size: batch size for experimental backend
 */
void set_tree_params(DecisionTreeParams &params, int cfg_max_depth,
                     int cfg_max_leaves, float cfg_max_features, int cfg_n_bins,
                     int cfg_min_samples_leaf, int cfg_min_samples_split,
                     float cfg_min_impurity_decrease,
                     CRITERION cfg_split_criterion, int cfg_max_batch_size) {
  params.max_depth = cfg_max_depth;
  params.max_leaves = cfg_max_leaves;
  params.max_features = cfg_max_features;
  params.n_bins = cfg_n_bins;
  params.min_samples_leaf = cfg_min_samples_leaf;
  params.min_samples_split = cfg_min_samples_split;
  params.split_criterion = cfg_split_criterion;
  params.min_impurity_decrease = cfg_min_impurity_decrease;
  params.max_batch_size = cfg_max_batch_size;
}

void validity_check(const DecisionTreeParams params) {
  ASSERT((params.max_depth >= 0), "Invalid max depth %d", params.max_depth);
  ASSERT((params.max_leaves == -1) || (params.max_leaves > 0),
         "Invalid max leaves %d", params.max_leaves);
  ASSERT((params.max_features > 0) && (params.max_features <= 1.0),
         "max_features value %f outside permitted (0, 1] range",
         params.max_features);
  ASSERT((params.n_bins > 0), "Invalid n_bins %d", params.n_bins);
  ASSERT((params.split_criterion != 3), "MAE not supported.");
  ASSERT((params.min_samples_leaf >= 1),
         "Invalid value for min_samples_leaf %d. Should be >= 1.",
         params.min_samples_leaf);
  ASSERT((params.min_samples_split >= 2),
         "Invalid value for min_samples_split: %d. Should be >= 2.",
         params.min_samples_split);
}

void print(const DecisionTreeParams params) {
  CUML_LOG_DEBUG("max_depth: %d", params.max_depth);
  CUML_LOG_DEBUG("max_leaves: %d", params.max_leaves);
  CUML_LOG_DEBUG("max_features: %f", params.max_features);
  CUML_LOG_DEBUG("n_bins: %d", params.n_bins);
  CUML_LOG_DEBUG("min_samples_leaf: %d", params.min_samples_leaf);
  CUML_LOG_DEBUG("min_samples_split: %d", params.min_samples_split);
  CUML_LOG_DEBUG("split_criterion: %d", params.split_criterion);
  CUML_LOG_DEBUG("min_impurity_decrease: %f", params.min_impurity_decrease);
  CUML_LOG_DEBUG("max_batch_size: %d", params.max_batch_size);
}

template <class T, class L>
std::string get_tree_summary_text(const TreeMetaDataNode<T, L> *tree) {
  std::ostringstream oss;
  oss << " Decision Tree depth --> " << tree->depth_counter
      << " and n_leaves --> " << tree->leaf_counter << "\n"
      << " Tree Fitting - Overall time --> "
      << (tree->prepare_time + tree->train_time) << " s"
      << "\n"
      << "   - preparing for fit time: " << tree->prepare_time << " s"
      << "\n"
      << "   - tree growing time: " << tree->train_time << " s";
  return oss.str();
}

template <class T, class L>
std::string get_tree_text(const TreeMetaDataNode<T, L> *tree) {
  std::string summary = get_tree_summary_text<T, L>(tree);
  return summary + "\n" + get_node_text<T, L>("", tree->sparsetree, 0, false);
}

template <class T, class L>
std::string get_tree_json(const TreeMetaDataNode<T, L> *tree) {
  std::ostringstream oss;
  return get_node_json("", tree->sparsetree, 0);
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierF *&tree, float *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params,
                               uint64_t seed) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  auto quantile_size = tree_params.n_bins * ncols;
  MLCommon::device_buffer<float> global_quantiles_buffer(
    handle.get_device_allocator(), handle.get_stream(), quantile_size);
  DecisionTree::computeQuantiles(
    global_quantiles_buffer.data(), tree_params.n_bins, data, nrows, ncols,
    handle.get_device_allocator(), handle.get_stream());
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params, seed,
                     global_quantiles_buffer.data());
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierD *&tree, double *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params,
                               uint64_t seed) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();

  auto quantile_size = tree_params.n_bins * ncols;
  MLCommon::device_buffer<double> global_quantiles_buffer(
    handle.get_device_allocator(), handle.get_stream(), quantile_size);
  DecisionTree::computeQuantiles(
    global_quantiles_buffer.data(), tree_params.n_bins, data, nrows, ncols,
    handle.get_device_allocator(), handle.get_stream());
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params, seed,
                     global_quantiles_buffer.data());
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierF *tree,
                                   const float *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierD *tree,
                                   const double *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

// ----------------------------- Regression ----------------------------------- //

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorF *&tree, float *data,
                              const int ncols, const int nrows, float *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params,
                              uint64_t seed) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  auto quantile_size = tree_params.n_bins * ncols;
  MLCommon::device_buffer<float> global_quantiles(
    handle.get_device_allocator(), handle.get_stream(), quantile_size);
  DecisionTree::computeQuantiles(
    global_quantiles.data(), tree_params.n_bins, data, nrows, ncols,
    handle.get_device_allocator(), handle.get_stream());
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params, seed, global_quantiles.data());
}

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorD *&tree, double *data,
                              const int ncols, const int nrows, double *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params,
                              uint64_t seed) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();

  auto quantile_size = tree_params.n_bins * ncols;
  MLCommon::device_buffer<double> global_quantiles(
    handle.get_device_allocator(), handle.get_stream(), quantile_size);
  DecisionTree::computeQuantiles(
    global_quantiles.data(), tree_params.n_bins, data, nrows, ncols,
    handle.get_device_allocator(), handle.get_stream());
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params, seed, global_quantiles.data());
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorF *tree, const float *rows,
                                  const int n_rows, const int n_cols,
                                  float *predictions, int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorD *tree,
                                  const double *rows, const int n_rows,
                                  const int n_cols, double *predictions,
                                  int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

// Functions' specializations
template std::string get_tree_summary_text<float, int>(
  const TreeClassifierF *tree);
template std::string get_tree_summary_text<double, int>(
  const TreeClassifierD *tree);
template std::string get_tree_summary_text<float, float>(
  const TreeRegressorF *tree);
template std::string get_tree_summary_text<double, double>(
  const TreeRegressorD *tree);

template std::string get_tree_text<float, int>(const TreeClassifierF *tree);
template std::string get_tree_text<double, int>(const TreeClassifierD *tree);
template std::string get_tree_text<float, float>(const TreeRegressorF *tree);
template std::string get_tree_text<double, double>(const TreeRegressorD *tree);

template std::string get_tree_json<float, int>(const TreeClassifierF *tree);
template std::string get_tree_json<double, int>(const TreeClassifierD *tree);
template std::string get_tree_json<float, float>(const TreeRegressorF *tree);
template std::string get_tree_json<double, double>(const TreeRegressorD *tree);

}  // End namespace DecisionTree
}  //End namespace ML
