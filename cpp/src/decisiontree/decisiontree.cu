#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/tree/decisiontree.hpp>
#include <cuml/tree/flatnode.h>
#include <raft/handle.hpp>

#include "decisiontree.cuh"

namespace ML {
namespace DT {

void validity_check(const DecisionTreeParams params)
{
  ASSERT((params.max_depth >= 0), "Invalid max depth %d", params.max_depth);
  ASSERT((params.max_leaves == -1) || (params.max_leaves > 0),
         "Invalid max leaves %d",
         params.max_leaves);
  ASSERT((params.max_features > 0) && (params.max_features <= 1.0),
         "max_features value %f outside permitted (0, 1] range",
         params.max_features);
  ASSERT((params.max_n_bins > 0), "Invalid max_n_bins %d", params.max_n_bins);
  ASSERT((params.max_n_bins <= 1024), "max_n_bins should not be larger than 1024");
  ASSERT((params.split_criterion != 3), "MAE not supported.");
  ASSERT((params.min_samples_leaf >= 1),
         "Invalid value for min_samples_leaf %d. Should be >= 1.",
         params.min_samples_leaf);
  ASSERT((params.min_samples_split >= 2),
         "Invalid value for min_samples_split: %d. Should be >= 2.",
         params.min_samples_split);
}

/**
 * @brief Set all DecisionTreeParams members.
 * @param[in,out] params: update with tree parameters
 * @param[in] cfg_max_depth: maximum tree depth; default -1
 * @param[in] cfg_max_leaves: maximum leaves; default -1
 * @param[in] cfg_max_features: maximum number of features; default 1.0f
 * @param[in] cfg_max_n_bins: maximum number of bins; default 128
 * @param[in] cfg_min_samples_leaf: min. rows in each leaf node; default 1
 * @param[in] cfg_min_samples_split: min. rows needed to split an internal node;
 *            default 2
 * @param[in] cfg_split_criterion: split criterion; default CRITERION_END,
 *            i.e., GINI for classification or MSE for regression
 * @param[in] cfg_max_batch_size: batch size for experimental backend
 */
void set_tree_params(DecisionTreeParams& params,
                     int cfg_max_depth,
                     int cfg_max_leaves,
                     float cfg_max_features,
                     int cfg_max_n_bins,
                     int cfg_min_samples_leaf,
                     int cfg_min_samples_split,
                     float cfg_min_impurity_decrease,
                     CRITERION cfg_split_criterion,
                     int cfg_max_batch_size)
{
  params.max_depth             = cfg_max_depth;
  params.max_leaves            = cfg_max_leaves;
  params.max_features          = cfg_max_features;
  params.max_n_bins            = cfg_max_n_bins;
  params.min_samples_leaf      = cfg_min_samples_leaf;
  params.min_samples_split     = cfg_min_samples_split;
  params.split_criterion       = cfg_split_criterion;
  params.min_impurity_decrease = cfg_min_impurity_decrease;
  params.max_batch_size        = cfg_max_batch_size;
  validity_check(params);
}

template <class T, class L>
std::string get_tree_summary_text(const TreeMetaDataNode<T, L>* tree)
{
  std::ostringstream oss;
  oss << " Decision Tree depth --> " << tree->depth_counter << " and n_leaves --> "
      << tree->leaf_counter << "\n"
      << " Tree Fitting - Overall time --> " << tree->train_time << " milliseconds"
      << "\n";
  return oss.str();
}

template <class T, class L>
std::string get_tree_text(const TreeMetaDataNode<T, L>* tree)
{
  std::string summary = get_tree_summary_text<T, L>(tree);
  return summary + "\n" + get_node_text<T, L>("", tree, 0, false);
}

template <class T, class L>
std::string get_tree_json(const TreeMetaDataNode<T, L>* tree)
{
  std::ostringstream oss;
  return get_node_json("", tree, 0);
}

// Functions' specializations
template std::string get_tree_summary_text<float, int>(const TreeClassifierF* tree);
template std::string get_tree_summary_text<double, int>(const TreeClassifierD* tree);
template std::string get_tree_summary_text<float, float>(const TreeRegressorF* tree);
template std::string get_tree_summary_text<double, double>(const TreeRegressorD* tree);

template std::string get_tree_text<float, int>(const TreeClassifierF* tree);
template std::string get_tree_text<double, int>(const TreeClassifierD* tree);
template std::string get_tree_text<float, float>(const TreeRegressorF* tree);
template std::string get_tree_text<double, double>(const TreeRegressorD* tree);

template std::string get_tree_json<float, int>(const TreeClassifierF* tree);
template std::string get_tree_json<double, int>(const TreeClassifierD* tree);
template std::string get_tree_json<float, float>(const TreeRegressorF* tree);
template std::string get_tree_json<double, double>(const TreeRegressorD* tree);

}  // End namespace DT
}  // End namespace ML
