#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/tree/decisiontree.hpp>

#include <cuml/tree/flatnode.h>
#include "decisiontree_impl.cuh"

namespace ML {
namespace DecisionTree {

/**
 * @brief Set all DecisionTreeParams members.
 * @param[in,out] params: update with tree parameters
 * @param[in] cfg_max_depth: maximum tree depth; default -1
 * @param[in] cfg_max_leaves: maximum leaves; default -1
 * @param[in] cfg_max_features: maximum number of features; default 1.0f
 * @param[in] cfg_n_bins: number of bins; default 8
 * @param[in] cfg_split_algo: split algorithm; default SPLIT_ALGO::HIST
 * @param[in] cfg_min_samples_leaf: min. rows in each leaf node; default 1
 * @param[in] cfg_min_samples_split: min. rows needed to split an internal node;
 *            default 2
 * @param[in] cfg_bootstrap_features: bootstrapping for features; default false
 * @param[in] cfg_split_criterion: split criterion; default CRITERION_END,
 *            i.e., GINI for classification or MSE for regression
 * @param[in] cfg_quantile_per_tree: compute quantile per tree; default false
 * @param[in] cfg_use_experimental_backend: Switch to using experimental
              backend; default false
 * @param[in] cfg_max_batch_size: batch size for experimental backend
 */
void set_tree_params(DecisionTreeParams &params, int cfg_max_depth,
                     int cfg_max_leaves, float cfg_max_features, int cfg_n_bins,
                     int cfg_split_algo, int cfg_min_samples_leaf,
                     int cfg_min_samples_split, float cfg_min_impurity_decrease,
                     bool cfg_bootstrap_features, CRITERION cfg_split_criterion,
                     bool cfg_quantile_per_tree,
                     bool cfg_use_experimental_backend,
                     int cfg_max_batch_size) {
  if (cfg_use_experimental_backend) {
    if (cfg_split_algo != SPLIT_ALGO::GLOBAL_QUANTILE) {
      CUML_LOG_WARN(
        "Experimental backend does not yet support histogram split algorithm");
      CUML_LOG_WARN(
        "To use experimental backend set split_algo = 1 (GLOBAL_QUANTILE)");
      cfg_use_experimental_backend = false;
    }
    if (cfg_quantile_per_tree) {
      CUML_LOG_WARN(
        "Experimental backend does not yet support per tree quantile "
        "computation");
      CUML_LOG_WARN(
        "To use experimental backend set quantile_per_tree = false");
      cfg_use_experimental_backend = false;
    }
    if (!cfg_use_experimental_backend) {
      CUML_LOG_WARN(
        "Not using the experimental backend due to above mentioned reason(s)");
      CUML_LOG_WARN("Switching back to default backend");
    }
  }

  params.max_depth = cfg_max_depth;
  params.max_leaves = cfg_max_leaves;
  params.max_features = cfg_max_features;
  params.n_bins = cfg_n_bins;
  params.split_algo = cfg_split_algo;
  params.min_samples_leaf = cfg_min_samples_leaf;
  params.min_samples_split = cfg_min_samples_split;
  params.bootstrap_features = cfg_bootstrap_features;
  params.split_criterion = cfg_split_criterion;
  params.quantile_per_tree = cfg_quantile_per_tree;
  params.use_experimental_backend = cfg_use_experimental_backend;
  params.min_impurity_decrease = cfg_min_impurity_decrease;
  params.max_batch_size = cfg_max_batch_size;
}

void validity_check(const DecisionTreeParams params) {
  ASSERT((params.max_depth >= 0), "Invalid max depth %d", params.max_depth);
  ASSERT((params.max_leaves == -1) || (params.max_leaves > 0),
         "Invalid max leaves %d", params.max_leaves);
  ASSERT((params.max_features > 0) && (params.max_features <= 1.0),
         "max_features value %f outside permitted (0, 1] range",
         params.max_features);
  ASSERT((params.n_bins > 0), "Invalid n_bins %d", params.n_bins);
  ASSERT((params.split_algo >= 0) &&
           (params.split_algo < SPLIT_ALGO::SPLIT_ALGO_END),
         "split_algo value %d outside permitted [0, %d) range",
         params.split_algo, SPLIT_ALGO::SPLIT_ALGO_END);
  ASSERT((params.min_samples_leaf >= 1),
         "Invalid value for min_samples_leaf %d. Should be >= 1.",
         params.min_samples_leaf);
  ASSERT((params.min_samples_split >= 2),
         "Invalid value for min_samples_split: %d. Should be >= 2.",
         params.min_samples_split);
}

void print(const DecisionTreeParams params) {
  CUML_LOG_DEBUG("max_depth: %d", params.max_depth);
  CUML_LOG_DEBUG("max_leaves: %d", params.max_leaves);
  CUML_LOG_DEBUG("max_features: %f", params.max_features);
  CUML_LOG_DEBUG("n_bins: %d", params.n_bins);
  CUML_LOG_DEBUG("split_algo: %d", params.split_algo);
  CUML_LOG_DEBUG("min_samples_leaf: %d", params.min_samples_leaf);
  CUML_LOG_DEBUG("min_samples_split: %d", params.min_samples_split);
  CUML_LOG_DEBUG("bootstrap_features: %d", params.bootstrap_features);
  CUML_LOG_DEBUG("split_criterion: %d", params.split_criterion);
  CUML_LOG_DEBUG("quantile_per_tree: %d", params.quantile_per_tree);
  CUML_LOG_DEBUG("min_impurity_decrease: %f", params.min_impurity_decrease);
  CUML_LOG_DEBUG("use_experimental_backend: %s",
                 params.use_experimental_backend ? "True" : "False");
  CUML_LOG_DEBUG("max_batch_size: %d", params.max_batch_size);
}

template <class T, class L>
std::string get_tree_summary_text(const TreeMetaDataNode<T, L> *tree) {
  std::ostringstream oss;
  oss << " Decision Tree depth --> " << tree->depth_counter
      << " and n_leaves --> " << tree->leaf_counter << "\n"
      << " Tree Fitting - Overall time --> "
      << (tree->prepare_time + tree->train_time) << " s"
      << "\n"
      << "   - preparing for fit time: " << tree->prepare_time << " s"
      << "\n"
      << "   - tree growing time: " << tree->train_time << " s";
  return oss.str();
}

template <class T, class L>
std::string get_tree_text(const TreeMetaDataNode<T, L> *tree) {
  std::string summary = get_tree_summary_text<T, L>(tree);
  return summary + "\n" + get_node_text<T, L>("", tree->sparsetree, 0, false);
}

template <class T, class L>
std::string get_tree_json(const TreeMetaDataNode<T, L> *tree) {
  std::ostringstream oss;
  return get_node_json("", tree->sparsetree, 0);
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierF *&tree, float *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params,
                               uint64_t seed) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  std::unique_ptr<MLCommon::device_buffer<float>> global_quantiles_buffer =
    nullptr;
  float *global_quantiles = nullptr;

  if (tree_params.use_experimental_backend) {
    auto quantile_size = tree_params.n_bins * ncols;
    global_quantiles_buffer = std::make_unique<MLCommon::device_buffer<float>>(
      handle.get_device_allocator(), handle.get_stream(), quantile_size);
    global_quantiles = global_quantiles_buffer->data();
    DecisionTree::computeQuantiles(global_quantiles, tree_params.n_bins, data,
                                   nrows, ncols, handle.get_device_allocator(),
                                   handle.get_stream());
  }
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params, seed, global_quantiles);
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierD *&tree, double *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params,
                               uint64_t seed) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();
  std::unique_ptr<MLCommon::device_buffer<double>> global_quantiles_buffer =
    nullptr;
  double *global_quantiles = nullptr;

  if (tree_params.use_experimental_backend) {
    auto quantile_size = tree_params.n_bins * ncols;
    global_quantiles_buffer = std::make_unique<MLCommon::device_buffer<double>>(
      handle.get_device_allocator(), handle.get_stream(), quantile_size);
    global_quantiles = global_quantiles_buffer->data();
    DecisionTree::computeQuantiles(global_quantiles, tree_params.n_bins, data,
                                   nrows, ncols, handle.get_device_allocator(),
                                   handle.get_stream());
  }
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params, seed, global_quantiles);
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierF *tree,
                                   const float *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierD *tree,
                                   const double *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

// ----------------------------- Regression ----------------------------------- //

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorF *&tree, float *data,
                              const int ncols, const int nrows, float *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params,
                              uint64_t seed) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  std::unique_ptr<MLCommon::device_buffer<float>> global_quantiles_buffer =
    nullptr;
  float *global_quantiles = nullptr;

  if (tree_params.use_experimental_backend) {
    auto quantile_size = tree_params.n_bins * ncols;
    global_quantiles_buffer = std::make_unique<MLCommon::device_buffer<float>>(
      handle.get_device_allocator(), handle.get_stream(), quantile_size);
    global_quantiles = global_quantiles_buffer->data();
    DecisionTree::computeQuantiles(global_quantiles, tree_params.n_bins, data,
                                   nrows, ncols, handle.get_device_allocator(),
                                   handle.get_stream());
  }
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params, seed, global_quantiles);
}

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorD *&tree, double *data,
                              const int ncols, const int nrows, double *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params,
                              uint64_t seed) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();
  std::unique_ptr<MLCommon::device_buffer<double>> global_quantiles_buffer =
    nullptr;
  double *global_quantiles = nullptr;

  if (tree_params.use_experimental_backend) {
    auto quantile_size = tree_params.n_bins * ncols;
    global_quantiles_buffer = std::make_unique<MLCommon::device_buffer<double>>(
      handle.get_device_allocator(), handle.get_stream(), quantile_size);
    global_quantiles = global_quantiles_buffer->data();
    DecisionTree::computeQuantiles(global_quantiles, tree_params.n_bins, data,
                                   nrows, ncols, handle.get_device_allocator(),
                                   handle.get_stream());
  }
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params, seed, global_quantiles);
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorF *tree, const float *rows,
                                  const int n_rows, const int n_cols,
                                  float *predictions, int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorD *tree,
                                  const double *rows, const int n_rows,
                                  const int n_cols, double *predictions,
                                  int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

// Functions' specializations
template std::string get_tree_summary_text<float, int>(
  const TreeClassifierF *tree);
template std::string get_tree_summary_text<double, int>(
  const TreeClassifierD *tree);
template std::string get_tree_summary_text<float, float>(
  const TreeRegressorF *tree);
template std::string get_tree_summary_text<double, double>(
  const TreeRegressorD *tree);

template std::string get_tree_text<float, int>(const TreeClassifierF *tree);
template std::string get_tree_text<double, int>(const TreeClassifierD *tree);
template std::string get_tree_text<float, float>(const TreeRegressorF *tree);
template std::string get_tree_text<double, double>(const TreeRegressorD *tree);

template std::string get_tree_json<float, int>(const TreeClassifierF *tree);
template std::string get_tree_json<double, int>(const TreeClassifierD *tree);
template std::string get_tree_json<float, float>(const TreeRegressorF *tree);
template std::string get_tree_json<double, double>(const TreeRegressorD *tree);

}  // End namespace DecisionTree
}  //End namespace ML
