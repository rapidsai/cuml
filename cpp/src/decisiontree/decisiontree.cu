#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/tree/flatnode.h>
#include <cuml/tree/decisiontree.hpp>
#include "decisiontree_impl.cuh"

namespace ML {
namespace DecisionTree {

void set_tree_params(DecisionTreeParams &params, int cfg_max_depth,
                     int cfg_max_leaves, float cfg_max_features, int cfg_n_bins,
                     int cfg_split_algo, int cfg_min_rows_per_node,
                     float cfg_min_impurity_decrease,
                     bool cfg_bootstrap_features, CRITERION cfg_split_criterion,
                     bool cfg_quantile_per_tree, bool cfg_shuffle_features) {
  params.max_depth = cfg_max_depth;
  params.max_leaves = cfg_max_leaves;
  params.max_features = cfg_max_features;
  params.n_bins = cfg_n_bins;
  params.split_algo = cfg_split_algo;
  params.min_rows_per_node = cfg_min_rows_per_node;
  params.bootstrap_features = cfg_bootstrap_features;
  params.split_criterion = cfg_split_criterion;
  params.quantile_per_tree = cfg_quantile_per_tree;
  params.shuffle_features = cfg_shuffle_features;
  params.min_impurity_decrease = cfg_min_impurity_decrease;
}

void validity_check(const DecisionTreeParams params) {
  ASSERT((params.max_depth >= 0), "Invalid max depth %d", params.max_depth);
  ASSERT((params.max_leaves == -1) || (params.max_leaves > 0),
         "Invalid max leaves %d", params.max_leaves);
  ASSERT((params.max_features > 0) && (params.max_features <= 1.0),
         "max_features value %f outside permitted (0, 1] range",
         params.max_features);
  ASSERT((params.n_bins > 0), "Invalid n_bins %d", params.n_bins);
  ASSERT((params.split_algo >= 0) &&
           (params.split_algo < SPLIT_ALGO::SPLIT_ALGO_END),
         "split_algo value %d outside permitted [0, %d) range",
         params.split_algo, SPLIT_ALGO::SPLIT_ALGO_END);
  ASSERT((params.min_rows_per_node >= 2),
         "Invalid min # rows per node value %d. Should be >= 2.",
         params.min_rows_per_node);
}

void print(const DecisionTreeParams params) {
  CUML_LOG_DEBUG("max_depth: %d", params.max_depth);
  CUML_LOG_DEBUG("max_leaves: %d", params.max_leaves);
  CUML_LOG_DEBUG("max_features: %f", params.max_features);
  CUML_LOG_DEBUG("n_bins: %d", params.n_bins);
  CUML_LOG_DEBUG("split_algo: %d", params.split_algo);
  CUML_LOG_DEBUG("min_rows_per_node: %d", params.min_rows_per_node);
  CUML_LOG_DEBUG("bootstrap_features: %d", params.bootstrap_features);
  CUML_LOG_DEBUG("split_criterion: %d", params.split_criterion);
  CUML_LOG_DEBUG("quantile_per_tree: %d", params.quantile_per_tree);
  CUML_LOG_DEBUG("shuffle_features: %d", params.shuffle_features);
}

template <class T, class L>
void print_tree_summary(const TreeMetaDataNode<T, L> *tree) {
  CUML_LOG_INFO(" Decision Tree depth --> %d and n_leaves --> %d",
                tree->depth_counter, tree->leaf_counter);
  CUML_LOG_INFO(" Tree Fitting - Overall time --> %lf s",
                tree->prepare_time + tree->train_time);
  CUML_LOG_INFO("   - preparing for fit time: %lf s", tree->prepare_time);
  CUML_LOG_INFO("   - tree growing time: %lf s", tree->train_time);
}

template <class T, class L>
void print_tree(const TreeMetaDataNode<T, L> *tree) {
  print_tree_summary<T, L>(tree);
  print_node<T, L>("", tree->sparsetree, 0, false);
}

template <class T, class L>
std::string dump_tree_as_json(const TreeMetaDataNode<T, L> *tree) {
  std::ostringstream oss;
  return dump_node_as_json("", tree->sparsetree, 0);
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierF *&tree, float *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params);
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierD *&tree, double *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params);
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierF *tree,
                                   const float *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierD *tree,
                                   const double *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

// ----------------------------- Regression ----------------------------------- //

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorF *&tree, float *data,
                              const int ncols, const int nrows, float *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params);
}

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorD *&tree, double *data,
                              const int ncols, const int nrows, double *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params);
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorF *tree, const float *rows,
                                  const int n_rows, const int n_cols,
                                  float *predictions, int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorD *tree,
                                  const double *rows, const int n_rows,
                                  const int n_cols, double *predictions,
                                  int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

// Functions' specializations
template void print_tree_summary<float, int>(const TreeClassifierF *tree);
template void print_tree_summary<double, int>(const TreeClassifierD *tree);
template void print_tree_summary<float, float>(const TreeRegressorF *tree);
template void print_tree_summary<double, double>(const TreeRegressorD *tree);

template void print_tree<float, int>(const TreeClassifierF *tree);
template void print_tree<double, int>(const TreeClassifierD *tree);
template void print_tree<float, float>(const TreeRegressorF *tree);
template void print_tree<double, double>(const TreeRegressorD *tree);

template std::string dump_tree_as_json<float, int>(const TreeClassifierF *tree);
template std::string dump_tree_as_json<double, int>(
  const TreeClassifierD *tree);
template std::string dump_tree_as_json<float, float>(
  const TreeRegressorF *tree);
template std::string dump_tree_as_json<double, double>(
  const TreeRegressorD *tree);

}  // End namespace DecisionTree
}  //End namespace ML
