#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/tree/flatnode.h>
#include <cuml/tree/decisiontree.hpp>
#include "decisiontree_impl.cuh"

namespace ML {
namespace DecisionTree {

/**
 * @brief Set all DecisionTreeParams members.
 * @param[in,out] params: update with tree parameters
 * @param[in] cfg_max_depth: maximum tree depth; default -1
 * @param[in] cfg_max_leaves: maximum leaves; default -1
 * @param[in] cfg_max_features: maximum number of features; default 1.0f
 * @param[in] cfg_n_bins: number of bins; default 8
 * @param[in] cfg_split_algo: split algorithm; default SPLIT_ALGO::HIST
 * @param[in] cfg_min_samples_leaf: min. rows in each leaf node; default 1
 * @param[in] cfg_min_samples_split: min. rows needed to split an internal node;
 *            default 2
 * @param[in] cfg_bootstrap_features: bootstrapping for features; default false
 * @param[in] cfg_split_criterion: split criterion; default CRITERION_END,
 *            i.e., GINI for classification or MSE for regression
 * @param[in] cfg_quantile_per_tree: compute quantile per tree; default false
 * @param[in] cfg_use_experimental_backend: Switch to using experimental
              backend; default false
 * @param[in] cfg_max_batch_size: batch size for experimental backend
 */
void set_tree_params(DecisionTreeParams &params, int cfg_max_depth,
                     int cfg_max_leaves, float cfg_max_features, int cfg_n_bins,
                     int cfg_split_algo, int cfg_min_samples_leaf,
                     int cfg_min_samples_split, float cfg_min_impurity_decrease,
                     bool cfg_bootstrap_features, CRITERION cfg_split_criterion,
                     bool cfg_quantile_per_tree,
                     bool cfg_use_experimental_backend,
                     int cfg_max_batch_size) {
  if (cfg_use_experimental_backend) {
    if (cfg_split_algo != SPLIT_ALGO::GLOBAL_QUANTILE) {
      CUML_LOG_WARN(
        "Experimental backend does not yet support histogram split algorithm");
      CUML_LOG_WARN(
        "To use experimental backend set split_algo = 1 (GLOBAL_QUANTILE)");
      cfg_use_experimental_backend = false;
    }
    if (cfg_max_features != 1.0) {
      CUML_LOG_WARN(
        "Experimental backend does not yet support feature sub-sampling");
      CUML_LOG_WARN("To use experimental backend set max_features = 1.0");
      cfg_use_experimental_backend = false;
    }
    if (cfg_quantile_per_tree) {
      CUML_LOG_WARN(
        "Experimental backend does not yet support per tree quantile "
        "computation");
      CUML_LOG_WARN(
        "To use experimental backend set quantile_per_tree = false");
      cfg_use_experimental_backend = false;
    }
    if (!cfg_use_experimental_backend) {
      CUML_LOG_WARN(
        "Not using the experimental backend due to above mentioned reason(s)");
      CUML_LOG_WARN("Switching back to default backend");
    }
  }

  params.max_depth = cfg_max_depth;
  params.max_leaves = cfg_max_leaves;
  params.max_features = cfg_max_features;
  params.n_bins = cfg_n_bins;
  params.split_algo = cfg_split_algo;
  params.min_samples_leaf = cfg_min_samples_leaf;
  params.min_samples_split = cfg_min_samples_split;
  params.bootstrap_features = cfg_bootstrap_features;
  params.split_criterion = cfg_split_criterion;
  params.quantile_per_tree = cfg_quantile_per_tree;
  params.use_experimental_backend = cfg_use_experimental_backend;
  params.min_impurity_decrease = cfg_min_impurity_decrease;
  params.max_batch_size = cfg_max_batch_size;
}

void validity_check(const DecisionTreeParams params) {
  ASSERT((params.max_depth >= 0), "Invalid max depth %d", params.max_depth);
  ASSERT((params.max_leaves == -1) || (params.max_leaves > 0),
         "Invalid max leaves %d", params.max_leaves);
  ASSERT((params.max_features > 0) && (params.max_features <= 1.0),
         "max_features value %f outside permitted (0, 1] range",
         params.max_features);
  ASSERT((params.n_bins > 0), "Invalid n_bins %d", params.n_bins);
  ASSERT((params.split_algo >= 0) &&
           (params.split_algo < SPLIT_ALGO::SPLIT_ALGO_END),
         "split_algo value %d outside permitted [0, %d) range",
         params.split_algo, SPLIT_ALGO::SPLIT_ALGO_END);
  ASSERT((params.min_samples_leaf >= 1),
         "Invalid value for min_samples_leaf %d. Should be >= 1.",
         params.min_samples_leaf);
  ASSERT((params.min_samples_split >= 2),
         "Invalid value for min_samples_split: %d. Should be >= 2.",
         params.min_samples_split);
}

void print(const DecisionTreeParams params) {
  CUML_LOG_DEBUG("max_depth: %d", params.max_depth);
  CUML_LOG_DEBUG("max_leaves: %d", params.max_leaves);
  CUML_LOG_DEBUG("max_features: %f", params.max_features);
  CUML_LOG_DEBUG("n_bins: %d", params.n_bins);
  CUML_LOG_DEBUG("split_algo: %d", params.split_algo);
  CUML_LOG_DEBUG("min_samples_leaf: %d", params.min_samples_leaf);
  CUML_LOG_DEBUG("min_samples_split: %d", params.min_samples_split);
  CUML_LOG_DEBUG("bootstrap_features: %d", params.bootstrap_features);
  CUML_LOG_DEBUG("split_criterion: %d", params.split_criterion);
  CUML_LOG_DEBUG("quantile_per_tree: %d", params.quantile_per_tree);
  CUML_LOG_DEBUG("min_impurity_decrease: %f", params.min_impurity_decrease);
  CUML_LOG_DEBUG("use_experimental_backend: %s",
                 params.use_experimental_backend ? "True" : "False");
  CUML_LOG_DEBUG("max_batch_size: %d", params.max_batch_size);
}

template <class T, class L>
void print_tree_summary(const TreeMetaDataNode<T, L> *tree) {
  CUML_LOG_INFO(" Decision Tree depth --> %d and n_leaves --> %d",
                tree->depth_counter, tree->leaf_counter);
  CUML_LOG_INFO(" Tree Fitting - Overall time --> %lf s",
                tree->prepare_time + tree->train_time);
  CUML_LOG_INFO("   - preparing for fit time: %lf s", tree->prepare_time);
  CUML_LOG_INFO("   - tree growing time: %lf s", tree->train_time);
}

template <class T, class L>
void print_tree(const TreeMetaDataNode<T, L> *tree) {
  print_tree_summary<T, L>(tree);
  print_node<T, L>("", tree->sparsetree, 0, false);
}

template <class T, class L>
std::string dump_tree_as_json(const TreeMetaDataNode<T, L> *tree) {
  std::ostringstream oss;
  return dump_node_as_json("", tree->sparsetree, 0);
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierF *&tree, float *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params);
}

void decisionTreeClassifierFit(const raft::handle_t &handle,
                               TreeClassifierD *&tree, double *data,
                               const int ncols, const int nrows, int *labels,
                               unsigned int *rowids, const int n_sampled_rows,
                               int unique_labels,
                               DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();
  dt_classifier->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                     unique_labels, tree, tree_params);
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierF *tree,
                                   const float *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<float>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<float>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

void decisionTreeClassifierPredict(const raft::handle_t &handle,
                                   const TreeClassifierD *tree,
                                   const double *rows, const int n_rows,
                                   const int n_cols, int *predictions,
                                   int verbosity) {
  std::shared_ptr<DecisionTreeClassifier<double>> dt_classifier =
    std::make_shared<DecisionTreeClassifier<double>>();
  dt_classifier->predict(handle, tree, rows, n_rows, n_cols, predictions,
                         verbosity);
}

// ----------------------------- Regression ----------------------------------- //

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorF *&tree, float *data,
                              const int ncols, const int nrows, float *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params);
}

void decisionTreeRegressorFit(const raft::handle_t &handle,
                              TreeRegressorD *&tree, double *data,
                              const int ncols, const int nrows, double *labels,
                              unsigned int *rowids, const int n_sampled_rows,
                              DecisionTree::DecisionTreeParams tree_params) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();
  dt_regressor->fit(handle, data, ncols, nrows, labels, rowids, n_sampled_rows,
                    tree, tree_params);
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorF *tree, const float *rows,
                                  const int n_rows, const int n_cols,
                                  float *predictions, int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<float>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<float>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

void decisionTreeRegressorPredict(const raft::handle_t &handle,
                                  const TreeRegressorD *tree,
                                  const double *rows, const int n_rows,
                                  const int n_cols, double *predictions,
                                  int verbosity) {
  std::shared_ptr<DecisionTreeRegressor<double>> dt_regressor =
    std::make_shared<DecisionTreeRegressor<double>>();
  dt_regressor->predict(handle, tree, rows, n_rows, n_cols, predictions,
                        verbosity);
}

// Functions' specializations
template void print_tree_summary<float, int>(const TreeClassifierF *tree);
template void print_tree_summary<double, int>(const TreeClassifierD *tree);
template void print_tree_summary<float, float>(const TreeRegressorF *tree);
template void print_tree_summary<double, double>(const TreeRegressorD *tree);

template void print_tree<float, int>(const TreeClassifierF *tree);
template void print_tree<double, int>(const TreeClassifierD *tree);
template void print_tree<float, float>(const TreeRegressorF *tree);
template void print_tree<double, double>(const TreeRegressorD *tree);

template std::string dump_tree_as_json<float, int>(const TreeClassifierF *tree);
template std::string dump_tree_as_json<double, int>(
  const TreeClassifierD *tree);
template std::string dump_tree_as_json<float, float>(
  const TreeRegressorF *tree);
template std::string dump_tree_as_json<double, double>(
  const TreeRegressorD *tree);

}  // End namespace DecisionTree
}  //End namespace ML
