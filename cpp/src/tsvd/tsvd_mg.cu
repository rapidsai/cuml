/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/cudart_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include <cuml/common/device_buffer.hpp>
#include <cuml/decomposition/sign_flip_mg.hpp>
#include <cuml/decomposition/tsvd.hpp>
#include <cuml/decomposition/tsvd_mg.hpp>
#include <opg/linalg/mm_aTa.hpp>
#include <opg/stats/mean.hpp>
#include <opg/stats/mean_center.hpp>
#include <opg/stats/stddev.hpp>
#include <raft/comms/comms.hpp>
#include <raft/cuda_utils.cuh>
#include <raft/linalg/eltwise.cuh>
#include <raft/matrix/math.cuh>
#include <raft/stats/mean_center.cuh>
#include "tsvd.cuh"

using namespace MLCommon;

namespace ML {
namespace TSVD {
namespace opg {

template <typename T>
void fit_impl(raft::handle_t &handle,
              std::vector<Matrix::Data<T> *> &input_data,
              Matrix::PartDescriptor &input_desc, T *components,
              T *singular_vals, paramsTSVD prms, hipStream_t *streams,
              int n_streams, bool verbose) {
  const auto &comm = handle.get_comms();
  hipblasHandle_t cublas_handle = handle.get_cublas_handle();
  const auto allocator = handle.get_device_allocator();

  // This variable should be updated to use `size_t`
  // Reference issue https://github.com/rapidsai/cuml/issues/2459
  int len = prms.n_cols * prms.n_cols;

  device_buffer<T> cov_data(allocator, streams[0], len);
  size_t cov_data_size = cov_data.size();
  Matrix::Data<T> cov{cov_data.data(), cov_data_size};

  LinAlg::opg::mm_aTa(handle, cov, input_data, input_desc, streams, n_streams);

  device_buffer<T> components_all(allocator, streams[0], len);
  device_buffer<T> explained_var_all(allocator, streams[0], prms.n_cols);

  ML::calEig(handle, cov.ptr, components_all.data(), explained_var_all.data(),
             prms, streams[0]);

  raft::matrix::truncZeroOrigin(components_all.data(), prms.n_cols, components,
                                prms.n_components, prms.n_cols, streams[0]);

  T scalar = T(1);
  raft::matrix::seqRoot(explained_var_all.data(), singular_vals, scalar,
                        prms.n_components, streams[0]);
}

/**
 * @brief performs MNMG fit operation for the tsvd
 * @input param handle: the internal cuml handle object
 * @input param rank_sizes: includes all the partition size information for the rank
 * @input param n_parts: number of partitions
 * @input param input: input data
 * @output param components: principal components of the input data
 * @output param singular_vals: singular values of the data
 * @input param prms: data structure that includes all the parameters from input size to algorithm
 * @input param verbose
 */
template <typename T>
void fit_impl(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
              size_t n_parts, Matrix::Data<T> **input, T *components,
              T *singular_vals, paramsTSVD prms, bool verbose) {
  int rank = handle.get_comms().get_rank();

  std::vector<Matrix::RankSizePair *> ranksAndSizes(rank_sizes,
                                                    rank_sizes + n_parts);

  std::vector<Matrix::Data<T> *> input_data(input, input + n_parts);
  Matrix::PartDescriptor input_desc(prms.n_rows, prms.n_cols, ranksAndSizes,
                                    rank);

  // TODO: These streams should come from raft::handle_t
  int n_streams = n_parts;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  fit_impl(handle, input_data, input_desc, components, singular_vals, prms,
           streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

template <typename T>
void transform_impl(raft::handle_t &handle,
                    std::vector<Matrix::Data<T> *> &input,
                    Matrix::PartDescriptor input_desc, T *components,
                    std::vector<Matrix::Data<T> *> &trans_input,
                    paramsTSVD prms, hipStream_t *streams, int n_streams,
                    bool verbose) {
  int rank = handle.get_comms().get_rank();

  hipblasHandle_t hipblas.h = handle.get_cublas_handle();
  const auto allocator = handle.get_device_allocator();

  std::vector<Matrix::RankSizePair *> local_blocks =
    input_desc.blocksOwnedBy(rank);

  for (int i = 0; i < input.size(); i++) {
    int si = i % n_streams;

    T alpha = T(1);
    T beta = T(0);
    raft::linalg::gemm(handle, input[i]->ptr, local_blocks[i]->size,
                       size_t(prms.n_cols), components, trans_input[i]->ptr,
                       local_blocks[i]->size, int(prms.n_components),
                       HIPBLAS_OP_N, HIPBLAS_OP_T, alpha, beta, streams[si]);
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }
}

/**
 * @brief performs MNMG transform operation for the tsvd.
 * @input param handle: the internal cuml handle object
 * @input param rank_sizes: includes all the partition size information for the rank
 * @input param n_parts: number of partitions
 * @input param input: input data
 * @input param components: principal components of the input data
 * @output param trans_input: transformed input data
 * @input param prms: data structure that includes all the parameters from input size to algorithm
 * @input param verbose
 */
template <typename T>
void transform_impl(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
                    size_t n_parts, Matrix::Data<T> **input, T *components,
                    Matrix::Data<T> **trans_input, paramsTSVD prms,
                    bool verbose) {
  int rank = handle.get_comms().get_rank();

  std::vector<Matrix::RankSizePair *> ranksAndSizes(rank_sizes,
                                                    rank_sizes + n_parts);
  std::vector<Matrix::Data<T> *> input_data(input, input + n_parts);
  Matrix::PartDescriptor input_desc(prms.n_rows, prms.n_cols, ranksAndSizes,
                                    rank);
  std::vector<Matrix::Data<T> *> trans_data(trans_input, trans_input + n_parts);

  // TODO: These streams should come from raft::handle_t
  int n_streams = n_parts;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  transform_impl(handle, input_data, input_desc, components, trans_data, prms,
                 streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

template <typename T>
void inverse_transform_impl(raft::handle_t &handle,
                            std::vector<Matrix::Data<T> *> &trans_input,
                            Matrix::PartDescriptor trans_input_desc,
                            T *components,
                            std::vector<Matrix::Data<T> *> &input,
                            paramsTSVD prms, hipStream_t *streams,
                            int n_streams, bool verbose) {
  hipblasHandle_t hipblas.h = handle.get_cublas_handle();
  const auto allocator = handle.get_device_allocator();
  std::vector<Matrix::RankSizePair *> local_blocks =
    trans_input_desc.partsToRanks;

  for (int i = 0; i < local_blocks.size(); i++) {
    int si = i % n_streams;
    T alpha = T(1);
    T beta = T(0);

    raft::linalg::gemm(handle, trans_input[i]->ptr, local_blocks[i]->size,
                       size_t(prms.n_components), components, input[i]->ptr,
                       local_blocks[i]->size, prms.n_cols, HIPBLAS_OP_N,
                       HIPBLAS_OP_N, alpha, beta, streams[si]);
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }
}

/**
 * @brief performs MNMG inverse transform operation for the output.
 * @input param handle: the internal cuml handle object
 * @input param rank_sizes: includes all the partition size information for the rank
 * @input param n_parts: number of partitions
 * @input param trans_input: transformed input data
 * @input param components: principal components of the input data
 * @output param input: input data
 * @input param prms: data structure that includes all the parameters from input size to algorithm
 * @input param verbose
 */
template <typename T>
void inverse_transform_impl(raft::handle_t &handle,
                            Matrix::RankSizePair **rank_sizes, size_t n_parts,
                            Matrix::Data<T> **trans_input, T *components,
                            Matrix::Data<T> **input, paramsTSVD prms,
                            bool verbose) {
  int rank = handle.get_comms().get_rank();

  std::vector<Matrix::RankSizePair *> ranksAndSizes(rank_sizes,
                                                    rank_sizes + n_parts);
  Matrix::PartDescriptor trans_desc(prms.n_rows, prms.n_components,
                                    ranksAndSizes, rank);
  std::vector<Matrix::Data<T> *> trans_data(trans_input, trans_input + n_parts);

  std::vector<Matrix::Data<T> *> input_data(input, input + n_parts);

  // TODO: These streams should come from raft::handle_t
  int n_streams = n_parts;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  inverse_transform_impl(handle, trans_data, trans_desc, components, input_data,
                         prms, streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

/**
 * @brief performs MNMG fit and transform operation for the tsvd.
 * @input param handle: the internal cuml handle object
 * @input param rank_sizes: includes all the partition size information for the rank
 * @input param n_parts: number of partitions
 * @input param input: input data
 * @output param trans_input: transformed input data
 * @output param components: principal components of the input data
 * @output param explained_var: explained var
 * @output param explained_var_ratio: the explained var ratio
 * @output param singular_vals: singular values of the data
 * @input param prms: data structure that includes all the parameters from input size to algorithm
 * @input param verbose
 */
template <typename T>
void fit_transform_impl(raft::handle_t &handle,
                        std::vector<Matrix::Data<T> *> &input_data,
                        Matrix::PartDescriptor &input_desc,
                        std::vector<Matrix::Data<T> *> &trans_data,
                        Matrix::PartDescriptor &trans_desc, T *components,
                        T *explained_var, T *explained_var_ratio,
                        T *singular_vals, paramsTSVD prms, bool verbose) {
  int rank = handle.get_comms().get_rank();

  // TODO: These streams should come from raft::handle_t
  int n_streams = input_desc.blocksOwnedBy(rank).size();
  ;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  fit_impl(handle, input_data, input_desc, components, singular_vals, prms,
           streams, n_streams, verbose);

  transform_impl(handle, input_data, input_desc, components, trans_data, prms,
                 streams, n_streams, verbose);

  PCA::opg::sign_flip(handle, trans_data, input_desc, components,
                      prms.n_components, streams, n_streams);

  device_buffer<T> mu_trans(handle.get_device_allocator(), streams[0],
                            prms.n_components);
  Matrix::Data<T> mu_trans_data{mu_trans.data(), size_t(prms.n_components)};

  Stats::opg::mean(handle, mu_trans_data, trans_data, trans_desc, streams,
                   n_streams);

  Matrix::Data<T> explained_var_data{explained_var, size_t(prms.n_components)};

  Stats::opg::var(handle, explained_var_data, trans_data, trans_desc,
                  mu_trans_data.ptr, streams, n_streams);

  device_buffer<T> mu(handle.get_device_allocator(), streams[0], prms.n_rows);
  Matrix::Data<T> mu_data{mu.data(), size_t(prms.n_rows)};

  Stats::opg::mean(handle, mu_data, input_data, input_desc, streams, n_streams);

  device_buffer<T> var_input(handle.get_device_allocator(), streams[0],
                             prms.n_rows);
  Matrix::Data<T> var_input_data{var_input.data(), size_t(prms.n_rows)};

  Stats::opg::var(handle, var_input_data, input_data, input_desc, mu_data.ptr,
                  streams, n_streams);

  device_buffer<T> total_vars(handle.get_device_allocator(), streams[0], 1);
  raft::stats::sum(total_vars.data(), var_input_data.ptr, 1, prms.n_cols, false,
                   streams[0]);

  T total_vars_h;
  raft::update_host(&total_vars_h, total_vars.data(), 1, streams[0]);
  CUDA_CHECK(hipStreamSynchronize(streams[0]));
  T scalar = T(1) / total_vars_h;

  raft::linalg::scalarMultiply(explained_var_ratio, explained_var, scalar,
                               prms.n_components, streams[0]);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

void fit(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
         size_t n_parts, Matrix::floatData_t **input, float *components,
         float *singular_vals, paramsTSVD prms, bool verbose) {
  fit_impl(handle, rank_sizes, n_parts, input, components, singular_vals, prms,
           verbose);
}

void fit(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
         size_t n_parts, Matrix::doubleData_t **input, double *components,
         double *singular_vals, paramsTSVD prms, bool verbose) {
  fit_impl(handle, rank_sizes, n_parts, input, components, singular_vals, prms,
           verbose);
}

void fit_transform(raft::handle_t &handle,
                   std::vector<Matrix::Data<float> *> &input_data,
                   Matrix::PartDescriptor &input_desc,
                   std::vector<Matrix::Data<float> *> &trans_data,
                   Matrix::PartDescriptor &trans_desc, float *components,
                   float *explained_var, float *explained_var_ratio,
                   float *singular_vals, paramsTSVD prms, bool verbose) {
  fit_transform_impl(handle, input_data, input_desc, trans_data, trans_desc,
                     components, explained_var, explained_var_ratio,
                     singular_vals, prms, verbose);
}

void fit_transform(raft::handle_t &handle,
                   std::vector<Matrix::Data<double> *> &input_data,
                   Matrix::PartDescriptor &input_desc,
                   std::vector<Matrix::Data<double> *> &trans_data,
                   Matrix::PartDescriptor &trans_desc, double *components,
                   double *explained_var, double *explained_var_ratio,
                   double *singular_vals, paramsTSVD prms, bool verbose) {
  fit_transform_impl(handle, input_data, input_desc, trans_data, trans_desc,
                     components, explained_var, explained_var_ratio,
                     singular_vals, prms, verbose);
}

void transform(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
               size_t n_parts, Matrix::Data<float> **input, float *components,
               Matrix::Data<float> **trans_input, paramsTSVD prms,
               bool verbose) {
  transform_impl(handle, rank_sizes, n_parts, input, components, trans_input,
                 prms, verbose);
}

void transform(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
               size_t n_parts, Matrix::Data<double> **input, double *components,
               Matrix::Data<double> **trans_input, paramsTSVD prms,
               bool verbose) {
  transform_impl(handle, rank_sizes, n_parts, input, components, trans_input,
                 prms, verbose);
}

void inverse_transform(raft::handle_t &handle,
                       Matrix::RankSizePair **rank_sizes, size_t n_parts,
                       Matrix::Data<float> **trans_input, float *components,
                       Matrix::Data<float> **input, paramsTSVD prms,
                       bool verbose) {
  inverse_transform_impl(handle, rank_sizes, n_parts, trans_input, components,
                         input, prms, verbose);
}

void inverse_transform(raft::handle_t &handle,
                       Matrix::RankSizePair **rank_sizes, size_t n_parts,
                       Matrix::Data<double> **trans_input, double *components,
                       Matrix::Data<double> **input, paramsTSVD prms,
                       bool verbose) {
  inverse_transform_impl(handle, rank_sizes, n_parts, trans_input, components,
                         input, prms, verbose);
}

}  // namespace opg
}  // namespace TSVD
}  // namespace ML
