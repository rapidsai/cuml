#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/linear_model/preprocess_mg.hpp>

#include <raft/cudart_utils.h>
#include <opg/linalg/norm.hpp>
#include <opg/matrix/math.hpp>
#include <opg/stats/mean.hpp>
#include <opg/stats/mean_center.hpp>
#include <raft/comms/comms.hpp>
#include <raft/cuda_utils.cuh>
#include <raft/linalg/gemm.cuh>
#include <raft/linalg/subtract.cuh>
#include <raft/matrix/math.cuh>
#include <rmm/device_uvector.hpp>

using namespace MLCommon;

namespace ML {
namespace GLM {
namespace opg {

template <typename T>
void preProcessData_impl(raft::handle_t &handle,
                         std::vector<Matrix::Data<T> *> &input_data,
                         Matrix::PartDescriptor &input_desc,
                         std::vector<Matrix::Data<T> *> &labels, T *mu_input,
                         T *mu_labels, T *norm2_input, bool fit_intercept,
                         bool normalize, hipStream_t *streams, int n_streams,
                         bool verbose) {
  const auto &comm = handle.get_comms();
  hipblasHandle_t cublas_handle = handle.get_cublas_handle();
  hipsolverHandle_t cusolver_handle = handle.get_cusolver_dn_handle();

  if (fit_intercept) {
    Matrix::Data<T> mu_input_data{mu_input, size_t(input_desc.N)};
    Stats::opg::mean(handle, mu_input_data, input_data, input_desc, streams,
                     n_streams);
    Stats::opg::mean_center(input_data, input_desc, mu_input_data, comm,
                            streams, n_streams);

    Matrix::PartDescriptor labels_desc = input_desc;
    labels_desc.N = size_t(1);

    Matrix::Data<T> mu_labels_data{mu_labels, size_t(1)};
    Stats::opg::mean(handle, mu_labels_data, labels, labels_desc, streams,
                     n_streams);
    Stats::opg::mean_center(labels, labels_desc, mu_labels_data, comm, streams,
                            n_streams);

    if (normalize) {
      Matrix::Data<T> norm2_input_data{norm2_input, size_t(input_desc.N)};
      LinAlg::opg::colNorm2(handle, norm2_input_data, input_data, input_desc,
                            streams, n_streams);

      Matrix::opg::matrixVectorBinaryDivSkipZero(
        input_data, input_desc, norm2_input_data, false, true, true, comm,
        streams, n_streams);
    }
  }
}

template <typename T>
void postProcessData_impl(raft::handle_t &handle,
                          std::vector<Matrix::Data<T> *> &input_data,
                          Matrix::PartDescriptor &input_desc,
                          std::vector<Matrix::Data<T> *> &labels, T *coef,
                          T *intercept, T *mu_input, T *mu_labels,
                          T *norm2_input, bool fit_intercept, bool normalize,
                          hipStream_t *streams, int n_streams, bool verbose) {
  const auto &comm = handle.get_comms();
  hipblasHandle_t cublas_handle = handle.get_cublas_handle();
  hipsolverHandle_t cusolver_handle = handle.get_cusolver_dn_handle();

  rmm::device_uvector<T> d_intercept(1, streams[0]);

  if (normalize) {
    Matrix::Data<T> norm2_input_data{norm2_input, input_desc.N};
    Matrix::opg::matrixVectorBinaryMult(input_data, input_desc,
                                        norm2_input_data, false, true, comm,
                                        streams, n_streams);
    raft::matrix::matrixVectorBinaryDivSkipZero(coef, norm2_input, size_t(1),
                                                input_desc.N, false, true,
                                                streams[0], true);
  }

  raft::linalg::gemm(handle, mu_input, 1, input_desc.N, coef,
                     d_intercept.data(), 1, 1, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     streams[0]);

  raft::linalg::subtract(d_intercept.data(), mu_labels, d_intercept.data(), 1,
                         streams[0]);
  raft::update_host(intercept, d_intercept.data(), 1, streams[0]);

  Matrix::Data<T> mu_input_data{mu_input, size_t(input_desc.N)};
  Stats::opg::mean_add(input_data, input_desc, mu_input_data, comm, streams,
                       n_streams);

  Matrix::PartDescriptor label_desc = input_desc;
  label_desc.N = size_t(1);
  Matrix::Data<T> mu_label_data{mu_labels, size_t(1)};
  Stats::opg::mean_add(labels, label_desc, mu_label_data, comm, streams,
                       n_streams);
}

void preProcessData(raft::handle_t &handle,
                    std::vector<Matrix::Data<float> *> &input_data,
                    Matrix::PartDescriptor &input_desc,
                    std::vector<Matrix::Data<float> *> &labels, float *mu_input,
                    float *mu_labels, float *norm2_input, bool fit_intercept,
                    bool normalize, hipStream_t *streams, int n_streams,
                    bool verbose) {
  preProcessData_impl(handle, input_data, input_desc, labels, mu_input,
                      mu_labels, norm2_input, fit_intercept, normalize, streams,
                      n_streams, verbose);
}

void preProcessData(raft::handle_t &handle,
                    std::vector<Matrix::Data<double> *> &input_data,
                    Matrix::PartDescriptor &input_desc,
                    std::vector<Matrix::Data<double> *> &labels,
                    double *mu_input, double *mu_labels, double *norm2_input,
                    bool fit_intercept, bool normalize, hipStream_t *streams,
                    int n_streams, bool verbose) {
  preProcessData_impl(handle, input_data, input_desc, labels, mu_input,
                      mu_labels, norm2_input, fit_intercept, normalize, streams,
                      n_streams, verbose);
}

void postProcessData(raft::handle_t &handle,
                     std::vector<Matrix::Data<float> *> &input_data,
                     Matrix::PartDescriptor &input_desc,
                     std::vector<Matrix::Data<float> *> &labels, float *coef,
                     float *intercept, float *mu_input, float *mu_labels,
                     float *norm2_input, bool fit_intercept, bool normalize,
                     hipStream_t *streams, int n_streams, bool verbose) {
  postProcessData_impl(handle, input_data, input_desc, labels, coef, intercept,
                       mu_input, mu_labels, norm2_input, fit_intercept,
                       normalize, streams, n_streams, verbose);
}

void postProcessData(raft::handle_t &handle,
                     std::vector<Matrix::Data<double> *> &input_data,
                     Matrix::PartDescriptor &input_desc,
                     std::vector<Matrix::Data<double> *> &labels, double *coef,
                     double *intercept, double *mu_input, double *mu_labels,
                     double *norm2_input, bool fit_intercept, bool normalize,
                     hipStream_t *streams, int n_streams, bool verbose) {
  postProcessData_impl(handle, input_data, input_desc, labels, coef, intercept,
                       mu_input, mu_labels, norm2_input, fit_intercept,
                       normalize, streams, n_streams, verbose);
}

}  // namespace opg
}  // namespace GLM
}  // namespace ML
