/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cuml/cuml.hpp>
#include <cuml/linear_model/glm.hpp>
#include "glm/qn/qn.h"
#include "ols.h"
#include "ridge.h"

namespace ML {
namespace GLM {

using namespace MLCommon;

void olsFit(const cumlHandle &handle, float *input, int n_rows, int n_cols,
            float *labels, float *coef, float *intercept, bool fit_intercept,
            bool normalize, int algo) {
  olsFit(handle.getImpl(), input, n_rows, n_cols, labels, coef, intercept,
         fit_intercept, normalize, handle.getStream(), algo);
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void olsFit(const cumlHandle &handle, double *input, int n_rows, int n_cols,
            double *labels, double *coef, double *intercept, bool fit_intercept,
            bool normalize, int algo) {
  olsFit(handle.getImpl(), input, n_rows, n_cols, labels, coef, intercept,
         fit_intercept, normalize, handle.getStream(), algo);
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void olsPredict(const cumlHandle &handle, const float *input, int n_rows,
                int n_cols, const float *coef, float intercept, float *preds) {
  olsPredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds,
             handle.getStream());
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void olsPredict(const cumlHandle &handle, const double *input, int n_rows,
                int n_cols, const double *coef, double intercept,
                double *preds) {
  olsPredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds,
             handle.getStream());
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgeFit(const cumlHandle &handle, float *input, int n_rows, int n_cols,
              float *labels, float *alpha, int n_alpha, float *coef,
              float *intercept, bool fit_intercept, bool normalize, int algo) {
  ridgeFit(handle.getImpl(), input, n_rows, n_cols, labels, alpha, n_alpha,
           coef, intercept, fit_intercept, normalize, handle.getStream(), algo);
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgeFit(const cumlHandle &handle, double *input, int n_rows, int n_cols,
              double *labels, double *alpha, int n_alpha, double *coef,
              double *intercept, bool fit_intercept, bool normalize, int algo) {
  ridgeFit(handle.getImpl(), input, n_rows, n_cols, labels, alpha, n_alpha,
           coef, intercept, fit_intercept, normalize, handle.getStream(), algo);
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgePredict(const cumlHandle &handle, const float *input, int n_rows,
                  int n_cols, const float *coef, float intercept,
                  float *preds) {
  ridgePredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds,
               handle.getStream());
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgePredict(const cumlHandle &handle, const double *input, int n_rows,
                  int n_cols, const double *coef, double intercept,
                  double *preds) {
  ridgePredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds,
               handle.getStream());
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void qnFit(const cumlHandle &cuml_handle, float *X, float *y, int N, int D,
           int C, bool fit_intercept, float l1, float l2, int max_iter,
           float grad_tol, int linesearch_max_iter, int lbfgs_memory,
           int verbosity, float *w0, float *f, int *num_iters, bool X_col_major,
           int loss_type) {
  qnFit(cuml_handle.getImpl(), X, y, N, D, C, fit_intercept, l1, l2, max_iter,
        grad_tol, linesearch_max_iter, lbfgs_memory, verbosity, w0, f,
        num_iters, X_col_major, loss_type, cuml_handle.getStream());
}

void qnFit(const cumlHandle &cuml_handle, double *X, double *y, int N, int D,
           int C, bool fit_intercept, double l1, double l2, int max_iter,
           double grad_tol, int linesearch_max_iter, int lbfgs_memory,
           int verbosity, double *w0, double *f, int *num_iters,
           bool X_col_major, int loss_type) {
  qnFit(cuml_handle.getImpl(), X, y, N, D, C, fit_intercept, l1, l2, max_iter,
        grad_tol, linesearch_max_iter, lbfgs_memory, verbosity, w0, f,
        num_iters, X_col_major, loss_type, cuml_handle.getStream());
}

void qnDecisionFunction(const cumlHandle &cuml_handle, float *X, int N, int D,
                        int C, bool fit_intercept, float *params,
                        bool X_col_major, int loss_type, float *preds) {
  qnDecisionFunction(cuml_handle.getImpl(), X, N, D, C, fit_intercept, params,
                     X_col_major, loss_type, preds, cuml_handle.getStream());
}

void qnDecisionFunction(const cumlHandle &cuml_handle, double *X, int N, int D,
                        int C, bool fit_intercept, double *params,
                        bool X_col_major, int loss_type, double *scores) {
  qnDecisionFunction(cuml_handle.getImpl(), X, N, D, C, fit_intercept, params,
                     X_col_major, loss_type, scores, cuml_handle.getStream());
}

void qnPredict(const cumlHandle &cuml_handle, float *X, int N, int D, int C,
               bool fit_intercept, float *params, bool X_col_major,
               int loss_type, float *scores) {
  qnPredict(cuml_handle.getImpl(), X, N, D, C, fit_intercept, params,
            X_col_major, loss_type, scores, cuml_handle.getStream());
}

void qnPredict(const cumlHandle &cuml_handle, double *X, int N, int D, int C,
               bool fit_intercept, double *params, bool X_col_major,
               int loss_type, double *preds) {
  qnPredict(cuml_handle.getImpl(), X, N, D, C, fit_intercept, params,
            X_col_major, loss_type, preds, cuml_handle.getStream());
}

}  // namespace GLM
}  // namespace ML
