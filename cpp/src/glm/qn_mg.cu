#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "qn/mg/qn_mg.cuh"
#include "qn/mg/standardization.cuh"
#include "qn/simple_mat/dense.hpp"

#include <cuml/common/logger.hpp>
#include <cuml/linear_model/qn.h>
#include <cuml/linear_model/qn_mg.hpp>

#include <raft/core/comms.hpp>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/error.hpp>
#include <raft/core/handle.hpp>
#include <raft/label/classlabels.cuh>
#include <raft/linalg/divide.cuh>
#include <raft/linalg/matrix_vector.cuh>
#include <raft/linalg/matrix_vector_op.cuh>
#include <raft/linalg/sqrt.cuh>
#include <raft/matrix/math.hpp>
#include <raft/stats/mean_center.cuh>
#include <raft/stats/stddev.cuh>
#include <raft/stats/sum.cuh>
#include <raft/util/cudart_utils.hpp>

#include <hip/hip_runtime.h>

#include <vector>
using namespace MLCommon;

#include <cumlprims/opg/matrix/math.hpp>
#include <cumlprims/opg/stats/mean.hpp>
#include <cumlprims/opg/stats/mean_center.hpp>
#include <cumlprims/opg/stats/stddev.hpp>

namespace ML {
namespace GLM {
namespace opg {

template <typename T>
std::vector<T> distinct_mg(const raft::handle_t& handle, T* y, size_t n)
{
  hipStream_t stream              = handle.get_stream();
  raft::comms::comms_t const& comm = raft::resource::get_comms(handle);
  int rank                         = comm.get_rank();
  int n_ranks                      = comm.get_size();

  rmm::device_uvector<T> unique_y(0, stream);
  raft::label::getUniquelabels(unique_y, y, n, stream);

  rmm::device_uvector<size_t> recv_counts(n_ranks, stream);
  auto send_count = raft::make_device_scalar<size_t>(handle, unique_y.size());
  comm.allgather(send_count.data_handle(), recv_counts.data(), 1, stream);
  comm.sync_stream(stream);

  std::vector<size_t> recv_counts_host(n_ranks);
  raft::copy(recv_counts_host.data(), recv_counts.data(), n_ranks, stream);
  raft::resource::sync_stream(handle);

  std::vector<size_t> displs(n_ranks);
  size_t pos = 0;
  for (int i = 0; i < n_ranks; ++i) {
    displs[i] = pos;
    pos += recv_counts_host[i];
  }

  rmm::device_uvector<T> recv_buff(displs.back() + recv_counts_host.back(), stream);
  comm.allgatherv(
    unique_y.data(), recv_buff.data(), recv_counts_host.data(), displs.data(), stream);
  comm.sync_stream(stream);

  rmm::device_uvector<T> global_unique_y(0, stream);
  int n_distinct =
    raft::label::getUniquelabels(global_unique_y, recv_buff.data(), recv_buff.size(), stream);

  std::vector<T> global_unique_y_host(global_unique_y.size());
  raft::copy(global_unique_y_host.data(), global_unique_y.data(), global_unique_y.size(), stream);
  raft::resource::sync_stream(handle);

  return global_unique_y_host;
}

template <typename T>
void qnFit_impl(const raft::handle_t& handle,
                const qn_params& pams,
                T* X,
                bool X_col_major,
                bool standardization,
                T* y,
                size_t N,
                size_t D,
                size_t C,
                T* w0,
                T* f,
                int* num_iters,
                size_t n_samples,
                int rank,
                int n_ranks)
{
  auto X_simple = SimpleDenseMat<T>(X, N, D, X_col_major ? COL_MAJOR : ROW_MAJOR);

  rmm::device_uvector<T> mean_std_buff(4 * D, handle.get_stream());
  Standardizer<T>* std_obj = NULL;
  if (standardization) std_obj = new Standardizer(handle, X_simple, n_samples, mean_std_buff);

  ML::GLM::opg::qn_fit_x_mg(handle,
                            pams,
                            X_simple,
                            y,
                            C,
                            w0,
                            f,
                            num_iters,
                            n_samples,
                            rank,
                            n_ranks,
                            std_obj);  // ignore sample_weight, svr_eps

  if (standardization) {
    int n_targets = ML::GLM::detail::qn_is_classification(pams.loss) && C == 2 ? 1 : C;
    std_obj->adapt_model_for_linearFwd(handle, w0, n_targets, D, pams.fit_intercept);
    delete std_obj;
  }

  return;
}

template <typename T>
void qnFit_impl(raft::handle_t& handle,
                std::vector<Matrix::Data<T>*>& input_data,
                Matrix::PartDescriptor& input_desc,
                std::vector<Matrix::Data<T>*>& labels,
                T* coef,
                const qn_params& pams,
                bool X_col_major,
                bool standardization,
                int n_classes,
                T* f,
                int* num_iters)
{
  RAFT_EXPECTS(input_data.size() == 1,
               "qn_mg.cu currently does not accept more than one input matrix");
  RAFT_EXPECTS(labels.size() == input_data.size(), "labels size does not equal to input_data size");

  auto data_X = input_data[0];
  auto data_y = labels[0];

  size_t n_samples = 0;
  for (auto p : input_desc.partsToRanks) {
    n_samples += p->size;
  }

  auto stream = handle.get_stream();

  qnFit_impl<T>(handle,
                pams,
                data_X->ptr,
                X_col_major,
                standardization,
                data_y->ptr,
                input_desc.totalElementsOwnedBy(input_desc.rank),
                input_desc.N,
                n_classes,
                coef,
                f,
                num_iters,
                input_desc.M,
                input_desc.rank,
                input_desc.uniqueRanks().size());
}

template <typename T>
std::vector<T> getUniquelabelsMG(const raft::handle_t& handle,
                                 Matrix::PartDescriptor& input_desc,
                                 std::vector<Matrix::Data<T>*>& labels)
{
  RAFT_EXPECTS(labels.size() == 1,
               "getUniqueLabelsMG currently does not accept more than one data chunk");
  Matrix::Data<T>* data_y = labels[0];
  size_t n_rows           = input_desc.totalElementsOwnedBy(input_desc.rank);
  return distinct_mg<T>(handle, data_y->ptr, n_rows);
}

template std::vector<float> getUniquelabelsMG(const raft::handle_t& handle,
                                              Matrix::PartDescriptor& input_desc,
                                              std::vector<Matrix::Data<float>*>& labels);

template std::vector<double> getUniquelabelsMG(const raft::handle_t& handle,
                                               Matrix::PartDescriptor& input_desc,
                                               std::vector<Matrix::Data<double>*>& labels);

template <typename T>
void qnFit(raft::handle_t& handle,
           std::vector<Matrix::Data<T>*>& input_data,
           Matrix::PartDescriptor& input_desc,
           std::vector<Matrix::Data<T>*>& labels,
           T* coef,
           const qn_params& pams,
           bool X_col_major,
           bool standardization,
           int n_classes,
           T* f,
           int* num_iters)
{
  qnFit_impl<T>(handle,
                input_data,
                input_desc,
                labels,
                coef,
                pams,
                X_col_major,
                standardization,
                n_classes,
                f,
                num_iters);
}

template void qnFit(raft::handle_t& handle,
                    std::vector<Matrix::Data<float>*>& input_data,
                    Matrix::PartDescriptor& input_desc,
                    std::vector<Matrix::Data<float>*>& labels,
                    float* coef,
                    const qn_params& pams,
                    bool X_col_major,
                    bool standardization,
                    int n_classes,
                    float* f,
                    int* num_iters);

template void qnFit(raft::handle_t& handle,
                    std::vector<Matrix::Data<double>*>& input_data,
                    Matrix::PartDescriptor& input_desc,
                    std::vector<Matrix::Data<double>*>& labels,
                    double* coef,
                    const qn_params& pams,
                    bool X_col_major,
                    bool standardization,
                    int n_classes,
                    double* f,
                    int* num_iters);

template <typename T, typename I>
void qnFitSparse_impl(const raft::handle_t& handle,
                      const qn_params& pams,
                      T* X_values,
                      I* X_cols,
                      I* X_row_ids,
                      I X_nnz,
                      bool standardization,
                      T* y,
                      size_t N,
                      size_t D,
                      size_t C,
                      T* w0,
                      T* f,
                      int* num_iters,
                      size_t n_samples,
                      int rank,
                      int n_ranks)
{
  auto X_simple = SimpleSparseMat<T, I>(X_values, X_cols, X_row_ids, X_nnz, N, D);

  size_t vec_size = raft::alignTo<size_t>(sizeof(T) * D, ML::GLM::detail::qn_align);
  rmm::device_uvector<T> mean_std_buff(4 * vec_size, handle.get_stream());
  Standardizer<T>* std_obj = NULL;

  if (standardization)
    std_obj = new Standardizer(handle, X_simple, n_samples, mean_std_buff, vec_size);

  ML::GLM::opg::qn_fit_x_mg(handle,
                            pams,
                            X_simple,
                            y,
                            C,
                            w0,
                            f,
                            num_iters,
                            n_samples,
                            rank,
                            n_ranks,
                            std_obj);  // ignore sample_weight, svr_eps

  if (standardization) {
    int n_targets = ML::GLM::detail::qn_is_classification(pams.loss) && C == 2 ? 1 : C;
    std_obj->adapt_model_for_linearFwd(handle, w0, n_targets, D, pams.fit_intercept);
    delete std_obj;
  }

  return;
}

template <typename T, typename I = int>
void qnFitSparse(raft::handle_t& handle,
                 std::vector<Matrix::Data<T>*>& input_values,
                 I* input_cols,
                 I* input_row_ids,
                 I X_nnz,
                 Matrix::PartDescriptor& input_desc,
                 std::vector<Matrix::Data<T>*>& labels,
                 T* coef,
                 const qn_params& pams,
                 bool standardization,
                 int n_classes,
                 T* f,
                 int* num_iters)
{
  RAFT_EXPECTS(input_values.size() == 1,
               "qn_mg.cu currently does not accept more than one input matrix");

  auto data_input_values = input_values[0];
  auto data_y            = labels[0];

  qnFitSparse_impl(handle,
                   pams,
                   data_input_values->ptr,
                   input_cols,
                   input_row_ids,
                   X_nnz,
                   standardization,
                   data_y->ptr,
                   input_desc.totalElementsOwnedBy(input_desc.rank),
                   input_desc.N,
                   n_classes,
                   coef,
                   f,
                   num_iters,
                   input_desc.M,
                   input_desc.rank,
                   input_desc.uniqueRanks().size());
}

template void qnFitSparse<float, int>(raft::handle_t& handle,
                                      std::vector<Matrix::Data<float>*>& input_values,
                                      int* input_cols,
                                      int* input_row_ids,
                                      int X_nnz,
                                      Matrix::PartDescriptor& input_desc,
                                      std::vector<Matrix::Data<float>*>& labels,
                                      float* coef,
                                      const qn_params& pams,
                                      bool standardization,
                                      int n_classes,
                                      float* f,
                                      int* num_iters);

template void qnFitSparse<double, int>(raft::handle_t& handle,
                                       std::vector<Matrix::Data<double>*>& input_values,
                                       int* input_cols,
                                       int* input_row_ids,
                                       int X_nnz,
                                       Matrix::PartDescriptor& input_desc,
                                       std::vector<Matrix::Data<double>*>& labels,
                                       double* coef,
                                       const qn_params& pams,
                                       bool standardization,
                                       int n_classes,
                                       double* f,
                                       int* num_iters);

template void qnFitSparse<float, int64_t>(raft::handle_t& handle,
                                          std::vector<Matrix::Data<float>*>& input_values,
                                          int64_t* input_cols,
                                          int64_t* input_row_ids,
                                          int64_t X_nnz,
                                          Matrix::PartDescriptor& input_desc,
                                          std::vector<Matrix::Data<float>*>& labels,
                                          float* coef,
                                          const qn_params& pams,
                                          bool standardization,
                                          int n_classes,
                                          float* f,
                                          int* num_iters);

template void qnFitSparse<double, int64_t>(raft::handle_t& handle,
                                           std::vector<Matrix::Data<double>*>& input_values,
                                           int64_t* input_cols,
                                           int64_t* input_row_ids,
                                           int64_t X_nnz,
                                           Matrix::PartDescriptor& input_desc,
                                           std::vector<Matrix::Data<double>*>& labels,
                                           double* coef,
                                           const qn_params& pams,
                                           bool standardization,
                                           int n_classes,
                                           double* f,
                                           int* num_iters);

};  // namespace opg
};  // namespace GLM
};  // namespace ML
