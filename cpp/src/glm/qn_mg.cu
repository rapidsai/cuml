/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/common/logger.hpp>
#include <raft/util/cudart_utils.hpp>

#include "qn/mg/qn_mg.cuh"
#include "qn/simple_mat/dense.hpp"
#include <hip/hip_runtime.h>
#include <cuml/common/logger.hpp>
#include <cuml/linear_model/qn.h>
#include <cuml/linear_model/qn_mg.hpp>
#include <raft/core/comms.hpp>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/error.hpp>
#include <raft/core/handle.hpp>
#include <raft/label/classlabels.cuh>
#include <raft/util/cudart_utils.hpp>
using namespace MLCommon;

namespace ML {
namespace GLM {
namespace opg {

template <typename T>
int distinct(const raft::handle_t& handle, T* y, size_t n_rows, int rank)
{
  hipStream_t stream = handle.get_stream();

  rmm::device_uvector<T> unique_labels(0, stream);
  raft::label::getUniquelabels(unique_labels, y, n_rows, stream);
  std::cout << "rank " << rank
            << raft::arr2Str(unique_labels.data(), unique_labels.size(), " unique_labels ", stream)
            << std::endl;

  raft::comms::comms_t const& communicator = raft::resource::get_comms(handle);
  auto recv_size = raft::make_device_scalar<size_t>(handle, unique_labels.size());
  communicator.allreduce(
    recv_size.data_handle(), recv_size.data_handle(), 1, raft::comms::op_t::SUM, stream);
  communicator.sync_stream(stream);
  size_t cpu_recv_size;
  raft::copy(&cpu_recv_size, recv_size.data_handle(), 1, stream);
  raft::resource::sync_stream(handle);

  int send_count = unique_labels.size();
  std::cout << "rank " << rank << " cpu_recv_size " << cpu_recv_size
            << ", send_count: " << send_count << std::endl;
  rmm::device_uvector<T> recv_buffer(cpu_recv_size, stream);
  communicator.allgather(unique_labels.data(), recv_buffer.data(), unique_labels.size(), stream);
  communicator.sync_stream(stream);
  std::cout << "rank " << rank
            << raft::arr2Str(unique_labels.data(),
                             unique_labels.size(),
                             " after allgather unique_labels ",
                             stream)
            << std::endl;
  std::cout << "rank " << rank
            << raft::arr2Str(recv_buffer.data(), recv_buffer.size(), " recv_buffer : ", stream)
            << std::endl;

  rmm::device_uvector<T> global_unique_labels(0, stream);
  int num_distinct = raft::label::getUniquelabels(
    global_unique_labels, recv_buffer.data(), recv_buffer.size(), stream);
  std::cout << "rank " << rank
            << raft::arr2Str(global_unique_labels.data(),
                             global_unique_labels.size(),
                             " global_unique_labels: ",
                             stream)
            << std::endl;

  return num_distinct;
}

template <typename T>
int distinct_v(const raft::handle_t& handle, T* y, size_t n_rows)
{
  hipStream_t stream              = handle.get_stream();
  raft::comms::comms_t const& comm = raft::resource::get_comms(handle);
  int rank                         = comm.get_rank();
  int n_ranks                      = comm.get_size();

  rmm::device_uvector<T> unique_y(0, stream);
  raft::label::getUniquelabels(unique_y, y, n_rows, stream);
  std::cout << "rank " << rank
            << raft::arr2Str(unique_y.data(), unique_y.size(), " unique_labels ", stream)
            << std::endl;

  rmm::device_uvector<size_t> recv_counts(n_ranks, stream);
  auto send_count = raft::make_device_scalar<size_t>(handle, unique_y.size());
  comm.allgather(send_count.data_handle(), recv_counts.data(), 1, stream);
  comm.sync_stream(stream);
  std::cout << "rank " << rank
            << raft::arr2Str(recv_counts.data(), recv_counts.size(), " recv_counts ", stream)
            << std::endl;

  std::vector<size_t> recv_counts_host(n_ranks);
  raft::copy(recv_counts_host.data(), recv_counts.data(), n_ranks, stream);

  std::vector<size_t> displs(n_ranks);
  for (int i = 0, pos = 0; i < n_ranks; ++i) {
    displs[i] = pos;
    pos += recv_counts_host[i];
  }

  // std::cout << "rank " << rank << " displs ";
  // for (auto e : displs)
  //   std::cout << e << " ";
  // std::cout << std::endl;
  rmm::device_uvector<T> recv_buff(displs.back() + recv_counts_host.back(), stream);
  comm.allgatherv(
    unique_y.data(), recv_buff.data(), recv_counts_host.data(), displs.data(), stream);
  comm.sync_stream(stream);

  std::cout << "rank " << rank
            << raft::arr2Str(recv_buff.data(), recv_buff.size(), " recv_buff ", stream)
            << std::endl;

  rmm::device_uvector<T> global_unique_y(0, stream);

  int n_distinct =
    raft::label::getUniquelabels(global_unique_y, recv_buff.data(), recv_buff.size(), stream);
  std::cout << "rank " << rank
            << raft::arr2Str(
                 global_unique_y.data(), global_unique_y.size(), " global_unique_y ", stream)
            << std::endl;
  return n_distinct;
}

template <typename T>
void qnFit_impl(const raft::handle_t& handle,
                const qn_params& pams,
                T* X,
                bool X_col_major,
                T* y,
                size_t N,
                size_t D,
                size_t C,
                T* w0,
                T* f,
                int* num_iters,
                size_t n_samples,
                int rank,
                int n_ranks)
{
  switch (pams.loss) {
    case QN_LOSS_LOGISTIC: {
      RAFT_EXPECTS(
        C == 2,
        "qn_mg.cu: only the LOGISTIC loss is supported currently. The number of classes must be 2");
    } break;
    default: {
      RAFT_EXPECTS(false, "qn_mg.cu: unknown loss function type (id = %d).", pams.loss);
    }
  }

  auto X_simple = SimpleDenseMat<T>(X, N, D, X_col_major ? COL_MAJOR : ROW_MAJOR);

  ML::GLM::opg::qn_fit_x_mg(handle,
                            pams,
                            X_simple,
                            y,
                            C,
                            w0,
                            f,
                            num_iters,
                            n_samples,
                            rank,
                            n_ranks);  // ignore sample_weight, svr_eps
  return;
}

template <typename T>
void qnFit_impl(raft::handle_t& handle,
                std::vector<Matrix::Data<T>*>& input_data,
                Matrix::PartDescriptor& input_desc,
                std::vector<Matrix::Data<T>*>& labels,
                T* coef,
                const qn_params& pams,
                bool X_col_major,
                int n_classes,
                T* f,
                int* num_iters)
{
  RAFT_EXPECTS(input_data.size() == 1,
               "qn_mg.cu currently does not accept more than one input matrix");
  RAFT_EXPECTS(labels.size() == input_data.size(), "labels size does not equal to input_data size");

  auto data_X = input_data[0];
  auto data_y = labels[0];

  size_t n_samples = 0;
  for (auto p : input_desc.partsToRanks) {
    n_samples += p->size;
  }

  qnFit_impl<T>(handle,
                pams,
                data_X->ptr,
                X_col_major,
                data_y->ptr,
                input_desc.totalElementsOwnedBy(input_desc.rank),
                input_desc.N,
                n_classes,
                coef,
                f,
                num_iters,
                input_desc.M,
                input_desc.rank,
                input_desc.uniqueRanks().size());
}

int qnCalNumClasses(const raft::handle_t& handle,
                    Matrix::PartDescriptor& input_desc,
                    std::vector<Matrix::Data<float>*>& labels)
{
  RAFT_EXPECTS(labels.size() == 1, "distinct currently does not accept more than one data chunk");
  Matrix::Data<float>* data_y = labels[0];
  int n_rows                  = input_desc.totalElementsOwnedBy(input_desc.rank);

  std::cout << "rank " << input_desc.rank << ", input_desc.M " << input_desc.M
            << ", totalElementsOwned " << input_desc.totalElementsOwnedBy(input_desc.rank)
            << std::endl;
  //<< ", elements: " << raft::arr2Str(data_y->ptr, n_rows, " ", handle.get_stream())
  //<< std::endl;

  // return distinct<float>(handle, data_y->ptr, n_rows, input_desc.rank);
  return distinct_v<float>(handle, data_y->ptr, n_rows);
}

void qnFit(raft::handle_t& handle,
           std::vector<Matrix::Data<float>*>& input_data,
           Matrix::PartDescriptor& input_desc,
           std::vector<Matrix::Data<float>*>& labels,
           float* coef,
           const qn_params& pams,
           bool X_col_major,
           int n_classes,
           float* f,
           int* num_iters)
{
  qnFit_impl<float>(
    handle, input_data, input_desc, labels, coef, pams, X_col_major, n_classes, f, num_iters);
}

};  // namespace opg
};  // namespace GLM
};  // namespace ML
