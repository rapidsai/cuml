/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/common/cuml_allocator.hpp>
#include <cuml/linear_model/ols_mg.hpp>
#include <cuml/linear_model/preprocess_mg.hpp>
#include <opg/linalg/lstsq.hpp>
#include <opg/stats/mean.hpp>
#include <raft/comms/comms.hpp>
#include <raft/cuda_utils.cuh>
#include <raft/linalg/add.cuh>
#include <raft/linalg/gemm.cuh>
#include <raft/matrix/math.cuh>
#include <raft/matrix/matrix.cuh>
#include <rmm/device_uvector.hpp>

using namespace MLCommon;

namespace ML {
namespace OLS {
namespace opg {

template <typename T>
void fit_impl(raft::handle_t &handle,
              std::vector<Matrix::Data<T> *> &input_data,
              Matrix::PartDescriptor &input_desc,
              std::vector<Matrix::Data<T> *> &labels, T *coef, T *intercept,
              bool fit_intercept, bool normalize, int algo,
              hipStream_t *streams, int n_streams, bool verbose) {
  const auto &comm = handle.get_comms();
  hipblasHandle_t cublas_handle = handle.get_cublas_handle();
  hipsolverHandle_t cusolver_handle = handle.get_cusolver_dn_handle();

  rmm::device_uvector<T> mu_input(0, streams[0]);
  rmm::device_uvector<T> norm2_input(0, streams[0]);
  rmm::device_uvector<T> mu_labels(0, streams[0]);

  if (fit_intercept) {
    mu_input.resize(input_desc.N, streams[0]);
    mu_labels.resize(1, streams[0]);
    if (normalize) {
      norm2_input.resize(input_desc.N, streams[0]);
    }

    GLM::opg::preProcessData(handle, input_data, input_desc, labels,
                             mu_input.data(), mu_labels.data(),
                             norm2_input.data(), fit_intercept, normalize,
                             streams, n_streams, verbose);
  }

  if (algo == 0 || input_desc.N == 1) {
    ASSERT(false, "olsFit: no algorithm with this id has been implemented");
  } else if (algo == 1) {
    LinAlg::opg::lstsqEig(handle, input_data, input_desc, labels, coef, streams,
                          n_streams);
  } else {
    ASSERT(false, "olsFit: no algorithm with this id has been implemented");
  }

  if (fit_intercept) {
    GLM::opg::postProcessData(handle, input_data, input_desc, labels, coef,
                              intercept, mu_input.data(), mu_labels.data(),
                              norm2_input.data(), fit_intercept, normalize,
                              streams, n_streams, verbose);
  } else {
    *intercept = T(0);
  }
}

/**
 * @brief performs MNMG fit operation for the ols
 * @input param handle: the internal cuml handle object
 * @input param rank_sizes: includes all the partition size information for the rank
 * @input param n_parts: number of partitions
 * @input param input: input data
 * @input param labels: labels data
 * @output param coef: learned regression coefficients
 * @output param intercept: intercept value
 * @input param fit_intercept: fit intercept or not
 * @input param normalize: normalize the data or not
 * @input param verbose
 */
template <typename T>
void fit_impl(raft::handle_t &handle,
              std::vector<Matrix::Data<T> *> &input_data,
              Matrix::PartDescriptor &input_desc,
              std::vector<Matrix::Data<T> *> &labels, T *coef, T *intercept,
              bool fit_intercept, bool normalize, int algo, bool verbose) {
  int rank = handle.get_comms().get_rank();

  // TODO: These streams should come from raft::handle_t

  int n_streams = input_desc.blocksOwnedBy(rank).size();
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  fit_impl(handle, input_data, input_desc, labels, coef, intercept,
           fit_intercept, normalize, algo, streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

template <typename T>
void predict_impl(raft::handle_t &handle,
                  std::vector<Matrix::Data<T> *> &input_data,
                  Matrix::PartDescriptor &input_desc, T *coef, T intercept,
                  std::vector<Matrix::Data<T> *> &preds, hipStream_t *streams,
                  int n_streams, bool verbose) {
  std::vector<Matrix::RankSizePair *> local_blocks = input_desc.partsToRanks;
  T alpha = T(1);
  T beta = T(0);

  for (int i = 0; i < input_data.size(); i++) {
    int si = i % n_streams;
    raft::linalg::gemm(handle, input_data[i]->ptr, local_blocks[i]->size,
                       input_desc.N, coef, preds[i]->ptr, local_blocks[i]->size,
                       size_t(1), HIPBLAS_OP_N, HIPBLAS_OP_N, alpha, beta,
                       streams[si]);

    raft::linalg::addScalar(preds[i]->ptr, preds[i]->ptr, intercept,
                            local_blocks[i]->size, streams[si]);
  }
}

template <typename T>
void predict_impl(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
                  size_t n_parts, Matrix::Data<T> **input, size_t n_rows,
                  size_t n_cols, T *coef, T intercept, Matrix::Data<T> **preds,
                  bool verbose) {
  int rank = handle.get_comms().get_rank();

  std::vector<Matrix::RankSizePair *> ranksAndSizes(rank_sizes,
                                                    rank_sizes + n_parts);
  std::vector<Matrix::Data<T> *> input_data(input, input + n_parts);
  Matrix::PartDescriptor input_desc(n_rows, n_cols, ranksAndSizes, rank);
  std::vector<Matrix::Data<T> *> preds_data(preds, preds + n_parts);

  // TODO: These streams should come from raft::handle_t
  int n_streams = n_parts;
  hipStream_t streams[n_streams];
  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  predict_impl(handle, input_data, input_desc, coef, intercept, preds_data,
               streams, n_streams, verbose);

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }

  for (int i = 0; i < n_streams; i++) {
    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }
}

void fit(raft::handle_t &handle, std::vector<Matrix::Data<float> *> &input_data,
         Matrix::PartDescriptor &input_desc,
         std::vector<Matrix::Data<float> *> &labels, float *coef,
         float *intercept, bool fit_intercept, bool normalize, int algo,
         bool verbose) {
  fit_impl(handle, input_data, input_desc, labels, coef, intercept,
           fit_intercept, normalize, algo, verbose);
}

void fit(raft::handle_t &handle,
         std::vector<Matrix::Data<double> *> &input_data,
         Matrix::PartDescriptor &input_desc,
         std::vector<Matrix::Data<double> *> &labels, double *coef,
         double *intercept, bool fit_intercept, bool normalize, int algo,
         bool verbose) {
  fit_impl(handle, input_data, input_desc, labels, coef, intercept,
           fit_intercept, normalize, algo, verbose);
}

void predict(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
             size_t n_parts, Matrix::Data<float> **input, size_t n_rows,
             size_t n_cols, float *coef, float intercept,
             Matrix::Data<float> **preds, bool verbose) {
  predict_impl(handle, rank_sizes, n_parts, input, n_rows, n_cols, coef,
               intercept, preds, verbose);
}

void predict(raft::handle_t &handle, Matrix::RankSizePair **rank_sizes,
             size_t n_parts, Matrix::Data<double> **input, size_t n_rows,
             size_t n_cols, double *coef, double intercept,
             Matrix::Data<double> **preds, bool verbose) {
  predict_impl(handle, rank_sizes, n_parts, input, n_rows, n_cols, coef,
               intercept, preds, verbose);
}

}  // namespace opg
}  // namespace OLS
}  // namespace ML
