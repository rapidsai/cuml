/*
 * Copyright (c) 2021-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "umap.cuh"

namespace ML {
namespace UMAP {

void find_ab(const raft::handle_t& handle, UMAPParams* params)
{
  hipStream_t stream = handle.get_stream();
  UMAPAlgo::find_ab(params, stream);
}

std::unique_ptr<raft::sparse::COO<float, int>> get_graph(
  const raft::handle_t& handle,
  float* X,  // input matrix
  float* y,  // labels
  int n,
  int d,
  knn_indices_dense_t* knn_indices,  // precomputed indices
  float* knn_dists,                  // precomputed distances
  UMAPParams* params)
{
  if (dispatch_to_uint64_t(n, params->n_neighbors, params->n_components))
    return _get_graph<uint64_t>(handle, X, y, n, d, knn_indices, knn_dists, params);
  else
    return _get_graph<int>(handle, X, y, n, d, knn_indices, knn_dists, params);
}

void refine(const raft::handle_t& handle,
            float* X,
            int n,
            int d,
            raft::sparse::COO<float>* graph,
            UMAPParams* params,
            float* embeddings)
{
  if (dispatch_to_uint64_t(n, params->n_neighbors, params->n_components))
    _refine<uint64_t>(handle, X, n, d, graph, params, embeddings);
  else
    _refine<int>(handle, X, n, d, graph, params, embeddings);
}

void init_and_refine(const raft::handle_t& handle,
                     float* X,
                     int n,
                     int d,
                     raft::sparse::COO<float>* graph,
                     UMAPParams* params,
                     float* embeddings)
{
  if (dispatch_to_uint64_t(n, params->n_neighbors, params->n_components))
    _init_and_refine<uint64_t>(handle, X, n, d, graph, params, embeddings);
  else
    _init_and_refine<int>(handle, X, n, d, graph, params, embeddings);
}

void fit(const raft::handle_t& handle,
         float* X,
         float* y,
         int n,
         int d,
         knn_indices_dense_t* knn_indices,
         float* knn_dists,
         UMAPParams* params,
         float* embeddings,
         raft::sparse::COO<float, int>* graph)
{
  if (dispatch_to_uint64_t(n, params->n_neighbors, params->n_components))
    _fit<uint64_t>(handle, X, y, n, d, knn_indices, knn_dists, params, embeddings, graph);
  else
    _fit<int>(handle, X, y, n, d, knn_indices, knn_dists, params, embeddings, graph);
}

void fit_sparse(const raft::handle_t& handle,
                int* indptr,
                int* indices,
                float* data,
                size_t nnz,
                float* y,
                int n,
                int d,
                int* knn_indices,
                float* knn_dists,
                UMAPParams* params,
                float* embeddings,
                raft::sparse::COO<float, int>* graph)
{
  if (dispatch_to_uint64_t(n, params->n_neighbors, params->n_components))
    _fit_sparse<uint64_t>(handle,
                          indptr,
                          indices,
                          data,
                          nnz,
                          y,
                          n,
                          d,
                          knn_indices,
                          knn_dists,
                          params,
                          embeddings,
                          graph);
  else
    _fit_sparse<int>(handle,
                     indptr,
                     indices,
                     data,
                     nnz,
                     y,
                     n,
                     d,
                     knn_indices,
                     knn_dists,
                     params,
                     embeddings,
                     graph);
}

void transform(const raft::handle_t& handle,
               float* X,
               int n,
               int d,
               float* orig_X,
               int orig_n,
               float* embedding,
               int embedding_n,
               UMAPParams* params,
               float* transformed)
{
  if (dispatch_to_uint64_t(n, params->n_neighbors, params->n_components))
    _transform<uint64_t>(
      handle, X, n, d, orig_X, orig_n, embedding, embedding_n, params, transformed);
  else
    _transform<int>(handle, X, n, d, orig_X, orig_n, embedding, embedding_n, params, transformed);
}

void transform_sparse(const raft::handle_t& handle,
                      int* indptr,
                      int* indices,
                      float* data,
                      size_t nnz,
                      int n,
                      int d,
                      int* orig_x_indptr,
                      int* orig_x_indices,
                      float* orig_x_data,
                      size_t orig_nnz,
                      int orig_n,
                      float* embedding,
                      int embedding_n,
                      UMAPParams* params,
                      float* transformed)
{
  if (dispatch_to_uint64_t(n, params->n_neighbors, params->n_components))
    _transform_sparse<uint64_t>(handle,
                                indptr,
                                indices,
                                data,
                                nnz,
                                n,
                                d,
                                orig_x_indptr,
                                orig_x_indices,
                                orig_x_data,
                                orig_nnz,
                                orig_n,
                                embedding,
                                embedding_n,
                                params,
                                transformed);
  else
    _transform_sparse<int>(handle,
                           indptr,
                           indices,
                           data,
                           nnz,
                           n,
                           d,
                           orig_x_indptr,
                           orig_x_indices,
                           orig_x_data,
                           orig_nnz,
                           orig_n,
                           embedding,
                           embedding_n,
                           params,
                           transformed);
}

}  // namespace UMAP
}  // namespace ML
