#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file linear_svm.cuh
 * @brief Fit linear SVM.
 */

#include <iostream>
#include <random>

#include <hipblas.h>
#include <cuml/svm/svm_model.h>
#include <cuml/svm/svm_parameter.h>
#include <raft/linalg/cublas_wrappers.h>
#include <raft/linalg/gemv.h>
#include <raft/linalg/transpose.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
#include <common/nvtx.hpp>
#include <label/classlabels.cuh>
#include <matrix/kernelfactory.cuh>
#include <raft/cuda_utils.cuh>
#include <raft/linalg/map.cuh>
#include <raft/linalg/unary_op.cuh>
#include <raft/matrix/matrix.cuh>
#include <rmm/device_uvector.hpp>

#include <glm/ols.cuh>
#include <glm/qn/qn.cuh>

#include <cuml/svm/linear.hpp>

namespace ML {
namespace SVM {

namespace {

template <typename T>
__global__ void transpose(
  T* out, const T* in, const T* mul, const int nRows, const int nCols, const bool withBias)
{
  int nCols1 = withBias ? nCols + 1 : nCols;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nCols1; i += blockDim.x * gridDim.x) {
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < nRows; j += blockDim.y * gridDim.y) {
      out[i + j * nCols1] = mul[j] * (i == nCols ? T(1.0) : in[i * nRows + j]);
    }
  }
}

template <typename T, int BlockSize>
__global__ void mean(T* out, const T* in, const int nRows, const int nCols)
{
  typedef hipcub::BlockReduce<T, BlockSize> BlockSum;
  __shared__ typename BlockSum::TempStorage shm;
  int i = blockIdx.x;
  T t   = 0;
  T s   = 0;
  if (i < nCols) {
    for (int j = threadIdx.x; j < nRows; j += blockDim.x)
      t += in[i * nRows + j];
    s = BlockSum(shm).Sum(t);
  }
  if (threadIdx.x == 0) out[i] = s / T(nRows);
}

inline bool isRegression(LinearSVMParams::Loss loss)
{
  return loss == LinearSVMParams::EPSILON_INSENSITIVE ||
         loss == LinearSVMParams::SQUARED_EPSILON_INSENSITIVE;
}

template <typename T>
struct SignFun {
  const T H1_value;
  __device__ T operator()(const T x) const { return x == H1_value ? 1 : -1; }
};

template <typename T>
struct IndicatorFun {
  const T H1_value;
  __device__ T operator()(const T x) const { return T(x == H1_value); }
};

template <typename T>
void predict_linear(const raft::handle_t& handle,
                    const T* X,
                    const T* w,
                    const int nRows,
                    const int nCols,
                    T* out,
                    hipStream_t stream)
{
  raft::linalg::gemv(handle, X, nRows, nCols, w, out, false, stream);
  const T* p = w + nCols;
  raft::linalg::unaryOp(
    out, out, nRows, [p] __device__(T x) -> T { return x + *p; }, stream);
}

template <typename T>
void predict_indicator(const raft::handle_t& handle,
                       const T* X,
                       const T* w,
                       const int nRows,
                       const int nCols,
                       T* out,
                       hipStream_t stream)
{
  raft::linalg::gemv(handle, X, nRows, nCols, w, out, false, stream);
  const T* p = w + nCols;
  raft::linalg::unaryOp(
    out, out, nRows, [p] __device__(T x) -> T { return T((x + *p) > 0); }, stream);
}

template <typename T>
void predict_prob(const raft::handle_t& handle,
                  const T* X,
                  const T* w,
                  const T* probScale,
                  const int nRows,
                  const int nCols,
                  T* out,
                  hipStream_t stream)
{
  raft::linalg::gemv(handle, X, nRows, nCols, w, out, false, stream);
  const T* p = w + nCols;
  raft::linalg::unaryOp(
    out,
    out,
    nRows,
    [p, probScale] __device__(T x) -> T {
      T z = probScale[0] * (x + *p) + probScale[1];
      T t = raft::myExp(z < 0 ? z : -z);
      T q = 1 / (1 + t);
      return q * (z < 0 ? t : T(1.0));
    },
    stream);
}

template <typename T>
void predict_log_prob(const raft::handle_t& handle,
                      const T* X,
                      const T* w,
                      const T* probScale,
                      const int nRows,
                      const int nCols,
                      T* out,
                      hipStream_t stream)
{
  raft::linalg::gemv(handle, X, nRows, nCols, w, out, false, stream);
  const T* p = w + nCols;
  raft::linalg::unaryOp(
    out,
    out,
    nRows,
    [p, probScale] __device__(T x) -> T {
      T z = probScale[0] * (x + *p) + probScale[1];
      T t = -raft::myLog(1 + raft::myExp(z < 0 ? z : -z));
      return t + (z < 0 ? z : T(0));
    },
    stream);
}

};  // namespace

template <typename T>
LinearSVMModel<T>::LinearSVMModel(const raft::handle_t& handle,
                                  const LinearSVMParams params,
                                  const T* X,
                                  const int nRows,
                                  const int nCols,
                                  const T* y,
                                  const T* sampleWeight)
  : params(params),
    handle(handle),
    nRows(nRows),
    nCols(nCols),
    w(nCols + 1, handle.get_stream()),
    probScale(params.probability ? 2 : 0, handle.get_stream())
{
  ML::PUSH_RANGE("Trace::LinearSVMModel::fit");
  hipStream_t stream = handle.get_stream();
  mean<T, 256><<<dim3(w.size(), 1, 1), dim3(256, 1, 1), 0, stream>>>(w.data(), X, nRows, nCols);

  auto nCols1   = (params.fit_intercept && params.penalized_intercept) ? nCols + 1 : nCols;
  int num_iters = 0;
  T target;
  T iC = params.C > 0 ? (1.0 / params.C) : 1.0;

  T* X1 = (T*)X;
  rmm::device_uvector<T> X1Buf(0, stream);
  if (params.fit_intercept && params.penalized_intercept) {
    X1Buf.resize(nCols1 * nRows, stream);
    X1 = X1Buf.data();
    CUDA_CHECK(hipMemcpyAsync(X1, X, sizeof(T) * nCols * nRows, hipMemcpyDeviceToDevice, stream));
    thrust::device_ptr<T> p(X1 + nCols * nRows);
    thrust::fill(thrust::cuda::par.on(stream), p, p + nRows, 1.0);
  }

  T* y1 = (T*)y;
  rmm::device_uvector<T> y1Buf(0, stream);
  if (!isRegression(params.loss)) {
    y1Buf.resize(nRows, stream);
    y1 = y1Buf.data();
    raft::linalg::unaryOp(y1, y, nRows, SignFun<T>{T(params.H1_value)}, stream);
  }

  int qn_loss = 99;
  switch (params.loss) {
    case LinearSVMParams::HINGE: qn_loss = 3; break;
    case LinearSVMParams::SQUARED_HINGE: qn_loss = 4; break;
    case LinearSVMParams::EPSILON_INSENSITIVE: qn_loss = 5; break;
    case LinearSVMParams::SQUARED_EPSILON_INSENSITIVE: qn_loss = 6; break;
    default: break;
  }
  GLM::qnFit<T>(handle,
                X1,
                true,
                y1,
                nRows,
                nCols1,
                1,
                params.fit_intercept && !params.penalized_intercept,
                T(params.penalty == LinearSVMParams::L1 ? iC : 0.0),
                T(params.penalty == LinearSVMParams::L2 ? iC : 0.0),
                params.max_iter,
                T(params.grad_tol),
                T(params.change_tol),
                params.linesearch_max_iter,
                params.lbfgs_memory,
                params.verbose,
                w.data(),
                &target,
                &num_iters,
                qn_loss,
                stream,
                (T*)sampleWeight,
                T(params.svr_sensitivity));

  CUDA_CHECK(hipStreamSynchronize(stream));
  CUML_LOG_DEBUG(
    "LinearSVM finished fitting in %d iterations out of maximum %d.", num_iters, params.max_iter);

  ML::POP_RANGE();
  if (!params.probability) return;
  ML::PUSH_RANGE("Trace::LinearSVMModel::fit-probabilities");

  rmm::device_uvector<T> xwBuf(nRows, stream);
  T* xw = xwBuf.data();
  predict_linear(handle, X, w.data(), nRows, nCols, xw, stream);
  raft::linalg::unaryOp(y1, y, nRows, IndicatorFun<T>{T(params.H1_value)}, stream);

  GLM::qnFit<T>(handle,
                xw,
                true,
                y1,
                nRows,
                1 /* D = 1 for only one parameter besides bias */,
                2 /* C = 2 classes forced by LogisticLoss */,
                true /* bias is the second parameter to fit */,
                0,
                0,
                params.max_iter,
                T(params.grad_tol),
                T(params.change_tol),
                params.linesearch_max_iter,
                params.lbfgs_memory,
                params.verbose,
                probScale.data(),
                &target,
                &num_iters,
                0 /* logistic loss*/,
                stream,
                (T*)sampleWeight);
  CUML_LOG_DEBUG("LinearSVM finished fitting probabilities in %d iterations out of maximum %d.",
                 num_iters,
                 params.max_iter);

  ML::POP_RANGE();
}

template <typename T>
void LinearSVMModel<T>::predict(const T* X, const int nRows, const int nCols, T* out) const
{
  ASSERT(nCols == this->nCols,
         "Number of features passed to predict() must be the same as for fitting (%d != %d).",
         nCols,
         this->nCols);
  hipStream_t stream = handle.get_stream();
  if (isRegression(params.loss)) {
    predict_linear(handle, X, w.data(), nRows, nCols, out, stream);
  } else {
    predict_indicator(handle, X, w.data(), nRows, nCols, out, stream);
  }
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template <typename T>
void LinearSVMModel<T>::predict_proba(
  const T* X, const int nRows, const int nCols, const bool log, T* out) const
{
  ASSERT(nCols == this->nCols,
         "Number of features passed to predict() must be the same as for fitting (%d != %d).",
         nCols,
         this->nCols);
  ASSERT(!isRegression(params.loss),
         "Predicting probabilities is not available for the regression model");
  ASSERT(
    params.probability,
    "The model was not trained to output probabilities (LinearSVMParams.probability == false).");

  hipStream_t stream = handle.get_stream();
  if (log) {
    predict_log_prob(handle, X, w.data(), probScale.data(), nRows, nCols, out, stream);
  } else {
    predict_prob(handle, X, w.data(), probScale.data(), nRows, nCols, out, stream);
  }
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template class LinearSVMModel<float>;
template class LinearSVMModel<double>;

}  // namespace SVM
}  // namespace ML
