#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ml_utils.h"

#include <cuda_utils.h>
#include <limits.h>
#include <hipcub/hipcub.hpp>

namespace ML {
namespace SVM {

__global__ void set_unavailable(bool *available, int n_rows, const int *idx,
                                int n_selected) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n_selected) {
    available[idx[tid]] = false;
  }
}

__global__ void update_priority(int *new_priority, int n_selected,
                                const int *new_idx, int n_ws, const int *idx,
                                const int *priority) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n_selected) {
    int my_new_idx = new_idx[tid];
    // The working set size is limited (~1024 elements) so we just loop through it
    for (int i = 0; i < n_ws; i++) {
      if (idx[i] == my_new_idx) new_priority[tid] = priority[i] + 1;
    }
  }
}
}  // namespace SVM
}  // namespace ML
