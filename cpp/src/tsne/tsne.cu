#include "hip/hip_runtime.h"

#pragma once

#include "common/cumlHandle.hpp"

#include "tsne/tsne.h"

#include "hipblas.h"
#include "distances.h"
#include "kernels.h"
#include "utils.h"

#define TEST_NNZ 12021


namespace ML {
using namespace MLCommon;

void TSNE(const cumlHandle &handle, const float *X, float *Y, const int n,
          const int p, const int n_components = 2, int n_neighbors = 90,
          const float *distances_vector = NULL, const long *indices_vector = NULL,
          float *VAL_vector = NULL, const int *COL_vector = NULL, const int *ROW_vector = NULL,
          const float perplexity = 30.0f, const int perplexity_max_iter = 100,
          const int perplexity_tol = 1e-5,
          const float early_exaggeration = 12.0f,
          const int exaggeration_iter = 250, const float min_gain = 0.01f,
          const float eta = 500.0f, const int max_iter = 1000,
          const float pre_momentum = 0.8, const float post_momentum = 0.5,
          const long long seed = -1, const bool initialize_embeddings = false) {
  auto d_alloc = handle.getDeviceAllocator();

  hipStream_t stream = handle.getStream();

  assert(n > 0 && p > 0 && n_components > 0 && n_neighbors > 0);
  if (n_neighbors > n) n_neighbors = n;

  // Some preliminary intializations for cuBLAS and cuML
  DEBUG("[Info] Create cuBLAS and cuML handles.\n");
  const int k = n_components;
  hipblasHandle_t BLAS = handle.getImpl().getCublasHandle();

  const float neg2 = -2.0f, beta = 0.0f, one = 1.0f;



  // Get distances
  DEBUG("[Info] Get distances\n");
  float *distances =
      (float *)d_alloc->allocate(n * n_neighbors * sizeof(float), stream);
  long *indices =
    (long *)d_alloc->allocate(sizeof(long) * n * n_neighbors, stream);

  if (distances == NULL && indices == NULL) {
    get_distances(X, n, p, indices, distances, n_neighbors, stream);
  }
  else {
    MLCommon::updateDevice(distances, distances_vector, n * n_neighbors, stream);
    MLCommon::updateDevice(indices, indices_vector, n * n_neighbors, stream);

    std::cout << MLCommon::arr2Str(distances, 20, "Distances", stream) << std::endl;
    std::cout << MLCommon::arr2Str(indices, 20, "indices", stream) << std::endl;
  }



  normalize_distances(n, distances, n_neighbors, stream);
#if IF_DEBUG
    printf("[Info]  Normalized distances\n\n");
    std::cout << MLCommon::arr2Str(distances, 20, "Distances", stream) << std::endl;
#endif


  // Get perplexity
  DEBUG("[Info] Get perplexity\n");
  float *P = (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
  float P_sum = determine_sigmas(distances, P, perplexity, perplexity_max_iter,
                                 perplexity_tol, n, n_neighbors, stream);
  d_alloc->deallocate(distances, n * n_neighbors * sizeof(float), stream);
  DEBUG("[Info] P_sum = %f\n", P_sum);
#if IF_DEBUG
    printf("[Info]  Perplexity results\n\n");
    std::cout << MLCommon::arr2Str(P, 20, "Perplexity", stream) << std::endl;
#endif



  // Convert data to COO layout
  float *VAL;
  int *COL, *ROW;
  int NNZ;

  if (VAL_vector == NULL) {
    DEBUG("[Info] Convert to COO layout and symmetrize\n");
    COO_t<float> P_PT;
    symmetrize_perplexity(P, indices, &P_PT, n, n_neighbors, P_sum,
                          early_exaggeration, stream);
    CUDA_CHECK(hipPeekAtLastError());
    NNZ = P_PT.nnz;
    VAL = P_PT.vals;
    COL = P_PT.rows;
    ROW = P_PT.cols;
  }
  else {
    NNZ = TEST_NNZ;
    VAL = (float *)d_alloc->allocate(sizeof(float) * NNZ, stream);
    COL = (int *)d_alloc->allocate(sizeof(int) * NNZ, stream);
    ROW = (int *)d_alloc->allocate(sizeof(int) * NNZ, stream);
    MLCommon::updateDevice(VAL, VAL_vector, NNZ, stream);
    MLCommon::updateDevice(COL, COL_vector, NNZ, stream);
    MLCommon::updateDevice(ROW, ROW_vector, NNZ, stream);
  }

#if IF_DEBUG
    printf("[Info]  Symmetrized Perplexity results\n\n");
    std::cout << MLCommon::arr2Str(VAL, 20, "Perplexity", stream) << std::endl;

    printf("[Info]  COL\n\n");
    std::cout << MLCommon::arr2Str(COL, 20, "Perplexity", stream) << std::endl;

    printf("[Info]  RWW\n\n");
    std::cout << MLCommon::arr2Str(ROW, 20, "Perplexity", stream) << std::endl;
#endif



  // Allocate data [NOTICE all Fortran Contiguous]
  DEBUG("[Info] Malloc data and space\n");
  float *noise = (float *)d_alloc->allocate(sizeof(float) * n, stream);
  hipMemset(noise, 0, sizeof(float) * n);
  //random_vector(noise, -0.003f, 0.003f, n, stream, seed);
  CUDA_CHECK(hipPeekAtLastError());

  if (initialize_embeddings) {
    random_vector(Y, -0.1f, 0.1f, n * k, stream, seed);
    CUDA_CHECK(hipPeekAtLastError());
  }

  float *Q = (float *)d_alloc->allocate(sizeof(float) * n * n, stream);
  float *norm = (float *)d_alloc->allocate(sizeof(float) * n, stream);
  float *Q_sum = (float *)d_alloc->allocate(sizeof(float) * n, stream);
  double *sum = (double *)d_alloc->allocate(sizeof(double), stream);

  float *attract = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);
  float *repel = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);

  float *iY = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);
  float *gains = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);

  // Do gradient updates
  float momentum = pre_momentum;
  double Z;
  int error;

  DEBUG("[Info] Start iterations\n");
  for (int iter = 0; iter < max_iter; iter++) {

    if (iter == exaggeration_iter) {
      momentum = post_momentum;
      float div = 1.0f / early_exaggeration;
      thrust::transform(__STREAM__, VAL, VAL + NNZ, VAL, div * _1);
    }

    // Get norm(Y)
    get_norm(Y, norm, n, k, stream);
// #if IF_DEBUG
//     printf("[Info]  Norm(y)\n\n");
//     std::cout << MLCommon::arr2Str(norm, 20, "norm", stream) << std::endl;
//     std::cout << MLCommon::arr2Str(norm + n/2, 20, "norm", stream) << std::endl;
// #endif

    // Do -2 * (Y @ Y.T)
    if (error = hipblasSsyrk(BLAS, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k,
                            &neg2, Y, n, &beta, Q, n)) {
      DEBUG("[ERROR]  Error from BLAS = %d", error);
      break;
    }
#if IF_DEBUG
    printf("[Info]  -2Y @ Y.T\n\n");
    std::cout << MLCommon::arr2Str(Q, 20, "-2YYT", stream) << std::endl;
    std::cout << MLCommon::arr2Str(Q + n*n - n, 20, "-2YYT", stream) << std::endl;
#endif

    // Form T = 1 / (1+d)
    Z = form_t_distribution(Q, norm, n, Q_sum, sum, stream);
    CUDA_CHECK(hipPeekAtLastError());

    DEBUG("[Info] Z =  %lf iter = %d\n", Z, iter);
// #if IF_DEBUG
//     printf("[Info]  Q 1/(1+d)\n\n");
//     std::cout << MLCommon::arr2Str(Q, 20, "QQ", stream);
//     std::cout << MLCommon::arr2Str(Q + n*n - n, 20, "QQ", stream);
// #endif

    // Compute attractive forces with COO matrix
    attractive_forces(VAL, COL, ROW, Q, Y, attract, NNZ,
                      n, k, stream);
    CUDA_CHECK(hipPeekAtLastError());
#if IF_DEBUG
    printf("[Info]  Attractive forces\n\n");
    std::cout << MLCommon::arr2Str(attract, 20, "attract", stream);
    std::cout << MLCommon::arr2Str(attract + n, 20, "attract", stream);
#endif


    // Change Q to Q**2 for repulsion
    postprocess_Q(Q, Q_sum, n, stream);
// #if IF_DEBUG
//     printf("[Info]  Q**2\n\n");
//     std::cout << MLCommon::arr2Str(Q, 20, "Q**2", stream);
//     std::cout << MLCommon::arr2Str(Q + n*n - n, 20, "Q**2", stream);

//     printf("[Info]  Q_sum\n\n");
//     std::cout << MLCommon::arr2Str(Q_sum, 20, "Q_sum", stream);
//     std::cout << MLCommon::arr2Str(Q_sum + n/2, 20, "Q_sum", stream);
// #endif


    // Compute repel_1 = Q @ Y
    if (error = hipblasSsymm(BLAS, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, n,
                            k, &one, Q, n, Y, n, &beta, repel, n)) {
      DEBUG("[ERROR]  Error from BLAS = %d", error);
      break;
    }
    CUDA_CHECK(hipPeekAtLastError());
#if IF_DEBUG
    printf("[Info]  Q @ Y\n\n");
    std::cout << MLCommon::arr2Str(repel, 20, "Q @ Y", stream);
    std::cout << MLCommon::arr2Str(repel + n, 20, "Q @ Y", stream);
#endif


    // Repel_2 = mean contributions yi - yj
    // Repel = Repel_1 - Repel_2
    repel_minus_QY(repel, Q_sum, Y, n, k, stream);
    CUDA_CHECK(hipPeekAtLastError());
#if IF_DEBUG
    printf("[Info]  repel - mean @ Y\n\n");
    std::cout << MLCommon::arr2Str(repel, 20, "repel - mean @ Y", stream);
    std::cout << MLCommon::arr2Str(repel + n, 20, "repel - mean @ Y", stream);
#endif


    // Integrate forces with momentum
    apply_forces(attract, repel, Y, iY, noise, gains, n, k, Z, min_gain,
                 momentum, eta, stream);
    CUDA_CHECK(hipPeekAtLastError());

#if IF_DEBUG
    printf("@@@[%d]@@@[Info]  Y after integration\n\n", iter);
    std::cout << MLCommon::arr2Str(Y, 20, "Y", stream);
    std::cout << MLCommon::arr2Str(Y + n, 20, "Y", stream);

    // printf("[Info]  gains after integration\n\n");
    // std::cout << MLCommon::arr2Str(gains, 20, "gains", stream);
    // std::cout << MLCommon::arr2Str(gains + n, 20, "gains", stream);

    // printf("[Info]  iY after integration\n\n");
    // std::cout << MLCommon::arr2Str(iY, 20, "iY", stream);
    // std::cout << MLCommon::arr2Str(iY + n, 20, "iY", stream);
#endif


// #if IF_DEBUG
//     if (iter == 3) break;
// #endif
  }


#if not IF_DEBUG
  P_PT.destroy();
#else
  d_alloc->deallocate(VAL, sizeof(float) * NNZ, stream);
  d_alloc->deallocate(COL, sizeof(int) * NNZ, stream);
  d_alloc->deallocate(ROW, sizeof(int) * NNZ, stream);
#endif

  d_alloc->deallocate(noise, sizeof(float) * n, stream);

  d_alloc->deallocate(Q, sizeof(float) * n * n, stream);
  d_alloc->deallocate(norm, sizeof(float) * n, stream);
  d_alloc->deallocate(Q_sum, sizeof(float) * n, stream);
  d_alloc->deallocate(sum, sizeof(double), stream);

  d_alloc->deallocate(attract, sizeof(float) * n * k, stream);
  d_alloc->deallocate(repel, sizeof(float) * n * k, stream);

  d_alloc->deallocate(iY, sizeof(float) * n * k, stream);
  d_alloc->deallocate(gains, sizeof(float) * n * k, stream);
}
}  // namespace ML
