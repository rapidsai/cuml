#include "hip/hip_runtime.h"

#pragma once

#include "common/cumlHandle.hpp"

#include "tsne/tsne.h"

#include "hipblas.h"
#include "distances.h"
//#include "kernels.h"
#include "fast_kernels.h"
#include "utils.h"

#define TEST_NNZ 12021


namespace ML {
using namespace MLCommon;

void TSNE(const cumlHandle &handle, const float *X, float *Y, const int n,
			const int p, const int n_components = 2, int n_neighbors = 90,
			const float *distances_vector = NULL, const long *indices_vector = NULL,
			float *VAL_vector = NULL, const int *COL_vector = NULL, const int *ROW_vector = NULL,
			const float perplexity = 30.0f, const int perplexity_max_iter = 100,
			const int perplexity_tol = 1e-5,
			const float early_exaggeration = 12.0f,
			const int exaggeration_iter = 500, const float min_gain = 0.01f,
			const float eta = 500.0f, const int max_iter = 10000,
			const float pre_momentum = 0.8, const float post_momentum = 0.5,
			const long long seed = -1, const bool initialize_embeddings = false,
			const bool verbose = false, const char *method = "Fast") {

	auto d_alloc = handle.getDeviceAllocator();
	hipStream_t stream = handle.getStream();

	assert(n > 0 && p > 0 && n_components > 0 && n_neighbors > 0);
	if (n_neighbors > n) n_neighbors = n;

	// Some preliminary intializations for cuBLAS and cuML
	DEBUG("[Info] Create cuBLAS and cuML handles.\n");
	const int k = n_components;
	hipblasHandle_t BLAS = handle.getImpl().getCublasHandle();

	const float neg2 = -2.0f, beta = 0.0f, one = 1.0f;


	// Get distances
	DEBUG("[Info] Get distances\n");
	float *distances =
			(float *)d_alloc->allocate(n * n_neighbors * sizeof(float), stream);
	long *indices =
		(long *)d_alloc->allocate(sizeof(long) * n * n_neighbors, stream);

	if (distances == NULL && indices == NULL) {
		get_distances(X, n, p, indices, distances, n_neighbors, stream);
	}
	else {
		MLCommon::updateDevice(distances, distances_vector, n * n_neighbors, stream);
		MLCommon::updateDevice(indices, indices_vector, n * n_neighbors, stream);

		std::cout << MLCommon::arr2Str(distances, 20, "Distances", stream) << std::endl;
		std::cout << MLCommon::arr2Str(indices, 20, "indices", stream) << std::endl;
	}



	normalize_distances(n, distances, n_neighbors, stream);
#if IF_DEBUG
		printf("[Info]  Normalized distances\n\n");
		std::cout << MLCommon::arr2Str(distances, 20, "Distances", stream) << std::endl;
#endif


	// Get perplexity
	DEBUG("[Info] Get perplexity\n");
	float *P = (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
	float P_sum = determine_sigmas(distances, P, perplexity, perplexity_max_iter,
									perplexity_tol, n, n_neighbors, stream);
	d_alloc->deallocate(distances, n * n_neighbors * sizeof(float), stream);
	DEBUG("[Info] P_sum = %f\n", P_sum);
#if IF_DEBUG
		printf("[Info]  Perplexity results\n\n");
		std::cout << MLCommon::arr2Str(P, 20, "Perplexity", stream) << std::endl;
#endif



	// Convert data to COO layout
	float *VAL;
	int *COL, *ROW;
	int NNZ;

	if (VAL_vector == NULL) {
		DEBUG("[Info] Convert to COO layout and symmetrize\n");
		COO_t<float> P_PT;
		symmetrize_perplexity(P, indices, &P_PT, n, n_neighbors, P_sum,
													early_exaggeration, stream);
		CUDA_CHECK(hipPeekAtLastError());
		NNZ = P_PT.nnz;
		VAL = P_PT.vals;
		COL = P_PT.rows;
		ROW = P_PT.cols;
	}
	else {
		NNZ = TEST_NNZ;
		VAL = (float *)d_alloc->allocate(sizeof(float) * NNZ, stream);
		COL = (int *)d_alloc->allocate(sizeof(int) * NNZ, stream);
		ROW = (int *)d_alloc->allocate(sizeof(int) * NNZ, stream);
		MLCommon::updateDevice(VAL, VAL_vector, NNZ, stream);
		MLCommon::updateDevice(COL, COL_vector, NNZ, stream);
		MLCommon::updateDevice(ROW, ROW_vector, NNZ, stream);
	}

#if IF_DEBUG
		printf("[Info]  Symmetrized Perplexity results\n\n");
		std::cout << MLCommon::arr2Str(VAL, 20, "Perplexity", stream) << std::endl;

		printf("[Info]  COL\n\n");
		std::cout << MLCommon::arr2Str(COL, 20, "Perplexity", stream) << std::endl;

		printf("[Info]  RWW\n\n");
		std::cout << MLCommon::arr2Str(ROW, 20, "Perplexity", stream) << std::endl;
#endif



	// Allocate data [NOTICE all Fortran Contiguous]
	DEBUG("[Info] Malloc data and space\n");
	float *noise = (float *)d_alloc->allocate(sizeof(float) * n, stream);
	hipMemset(noise, 0, sizeof(float) * n);
	//random_vector(noise, -0.003f, 0.003f, n, stream, seed);
	CUDA_CHECK(hipPeekAtLastError());

	if (initialize_embeddings) {
		random_vector(Y, -0.1f, 0.1f, n * k, stream, seed);
		CUDA_CHECK(hipPeekAtLastError());
	}


	float *norm = (float *)d_alloc->allocate(sizeof(float) * n, stream);
	float *Q_sum = (float *)d_alloc->allocate(sizeof(float) * n, stream);
	double *sum = (double *)d_alloc->allocate(sizeof(double), stream);

	float *attract = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);
	float *repel = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);

	float *iY = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);
	float *gains = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);

	// Do gradient updates
	float momentum = pre_momentum;
	float Z;
	int error;

	if (method == "Fast") {

		for (int iter = 0; iter < max_iter; iter++) {
			if (iter == exaggeration_iter) {
				momentum = post_momentum;
				// Divide perplexities
				float div = 1.0f / early_exaggeration;
				thrust::transform(__STREAM__, VAL, VAL + NNZ, VAL, div * _1);
			}
			// Get norm(Y)
			get_norm(Y, norm, n, k, stream);

			// Fast compute attractive forces from COO matrix
			attractive_fast(VAL, COL, ROW, Y, norm, attract, NNZ, n, n_components, stream);

			// Fast compute repulsive forces
			Z = repulsive_fast(Y, repel, norm, Q_sum, n, n_components, stream);
			if (verbose) printf("[INFO]	Z at iter = %d is %lf", iter, Z);

			// Integrate forces with momentum
			apply_forces(attract, repel, Y, iY, noise, gains, n, k, Z, min_gain,
						momentum, eta, stream);
		}
	}

	else if (method == "Naive") {

		float *Q = (float *)d_alloc->allocate(sizeof(float) * n * n, stream);

		for (int iter = 0; iter < max_iter; iter++) {
			if (iter == exaggeration_iter) {
				momentum = post_momentum;
				// Divide perplexities
				float div = 1.0f / early_exaggeration;
				thrust::transform(__STREAM__, VAL, VAL + NNZ, VAL, div * _1);
			}
			// Get norm(Y)
			get_norm(Y, norm, n, k, stream);

			// Find Y @ Y.T
			if (error = hipblasSsyrk(BLAS, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k,
									&neg2, Y, n, &beta, Q, n)) {
				if (verbose) printf("[ERROR]	BLAS failed. Terminating TSNE\n");
				break;
			}
			CUDA_CHECK(hipPeekAtLastError());

			// Form T = 1 / (1 + d) = 1 / (1 + -2*Y@Y.T )
			Z = form_t_distribution(Q, norm, n, Q_sum, sum, stream);
			if (verbose) printf("[INFO]	Z at iter = %d is %lf", iter, Z);

			// Compute attractive forces from COO matrix
			attractive_forces(VAL, COL, ROW, Q, Y, attract, NNZ, n, k, stream);

			// Change Q to Q**2
			postprocess_Q(Q, Q_sum, n, stream);

			// Do Q**2 @ Y
			if (error = hipblasSsymm(BLAS, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, n,
			                        k, &one, Q, n, Y, n, &beta, repel, n)) {
				if (verbose) printf("[ERROR]	BLAS failed. Terminating TSNE\n");
				break;
			}
			CUDA_CHECK(hipPeekAtLastError());

			// Compute repel - Q**2 @ mean_Y
			repel_minus_QY(repel, Q_sum, Y, n, k, stream);

			// Integrate forces with momentum
			apply_forces(attract, repel, Y, iY, noise, gains, n, k, Z, min_gain,
						momentum, eta, stream);
		}

		d_alloc->deallocate(Q, sizeof(float) * n * n, stream);
	}


#if not IF_DEBUG
	P_PT.destroy();
#else
	d_alloc->deallocate(VAL, sizeof(float) * NNZ, stream);
	d_alloc->deallocate(COL, sizeof(int) * NNZ, stream);
	d_alloc->deallocate(ROW, sizeof(int) * NNZ, stream);
#endif

	d_alloc->deallocate(noise, sizeof(float) * n, stream);

	
	d_alloc->deallocate(norm, sizeof(float) * n, stream);
	d_alloc->deallocate(Q_sum, sizeof(float) * n, stream);
	d_alloc->deallocate(sum, sizeof(double), stream);

	d_alloc->deallocate(attract, sizeof(float) * n * k, stream);
	d_alloc->deallocate(repel, sizeof(float) * n * k, stream);

	d_alloc->deallocate(iY, sizeof(float) * n * k, stream);
	d_alloc->deallocate(gains, sizeof(float) * n * k, stream);
}
}  // namespace ML
