/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tsne_runner.cuh"
#include <cuml/manifold/tsne.h>

namespace ML {

template <typename tsne_input, typename value_idx, typename value_t>
value_t _fit(const raft::handle_t& handle,
             tsne_input& input,
             knn_graph<value_idx, value_t>& k_graph,
             TSNEParams& params)
{
  TSNE_runner<tsne_input, value_idx, value_t> runner(handle, input, k_graph, params);

  return runner.run();  // returns the Kullback–Leibler divergence
}

void TSNE_fit(const raft::handle_t& handle,
              float* X,
              float* Y,
              int n,
              int p,
              int64_t* knn_indices,
              float* knn_dists,
              TSNEParams& params,
              float* kl_div)
{
  ASSERT(n > 0 && p > 0 && params.dim > 0 && params.n_neighbors > 0 && X != NULL && Y != NULL,
         "Wrong input args");

  manifold_dense_inputs_t<float> input(X, Y, n, p);
  knn_graph<int64_t, float> k_graph(n, params.n_neighbors, knn_indices, knn_dists);

  float kl_div_v = _fit<manifold_dense_inputs_t<float>, knn_indices_dense_t, float>(
    handle, input, k_graph, params);

  if (kl_div) { *kl_div = kl_div_v; }
}

void TSNE_fit_sparse(const raft::handle_t& handle,
                     int* indptr,
                     int* indices,
                     float* data,
                     float* Y,
                     int nnz,
                     int n,
                     int p,
                     int* knn_indices,
                     float* knn_dists,
                     TSNEParams& params,
                     float* kl_div)
{
  ASSERT(n > 0 && p > 0 && params.dim > 0 && params.n_neighbors > 0 && indptr != NULL &&
           indices != NULL && data != NULL && Y != NULL,
         "Wrong input args");

  manifold_sparse_inputs_t<int, float> input(indptr, indices, data, Y, nnz, n, p);
  knn_graph<int, float> k_graph(n, params.n_neighbors, knn_indices, knn_dists);

  float kl_div_v = _fit<manifold_sparse_inputs_t<int, float>, knn_indices_sparse_t, float>(
    handle, input, k_graph, params);

  if (kl_div) { *kl_div = kl_div_v; }
}

}  // namespace ML
