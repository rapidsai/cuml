#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#pragma once

#include "common/cumlHandle.hpp"

#include "tsne/tsne.h"

#include "hipblas.h"
#include "distances.h"
#include "slow_kernels.h"
#include "fast_kernels.h"
#include "utils.h"


namespace ML {

void 
TSNE_fit(const cumlHandle &handle, const float *X, float *Y, const int n,
        const int p, const int n_components = 2, int n_neighbors = 30,
        const float perplexity = 30.0f, const int perplexity_max_iter = 100,
        const int perplexity_tol = 1e-5,
        const float early_exaggeration = 12.0f,
        const int exaggeration_iter = 150, const float min_gain = 0.01f,
        const float eta = 500.0f, const int max_iter = 500,
        const float pre_momentum = 0.8, const float post_momentum = 0.5,
        const long long seed = -1, const bool initialize_embeddings = true,
        const bool verbose = true)
    // Method = 0 for Naive, 1 for Fast
{
    assert(n > 0 && p > 0 && n_components > 0 && n_neighbors > 0 && X != NULL && Y != NULL);
    auto d_alloc = handle.getDeviceAllocator();
    hipStream_t stream = handle.getStream();

    if (n_neighbors > n) n_neighbors = n;

    // Notice perplexity must be <= than # of datapoints
    if (perplexity >= n) perplexity = n;
    if (verbose)
        printf("[Info]  Data = (%d, %d) with n_components = %d and perplexity = %f\n", n, p, n_components, perplexity);


    // Get distances
    if (verbose) printf("[Info] Getting distances.\n");

    float *distances = (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
    long *indices = (long *)d_alloc->allocate(sizeof(long) * n * n_neighbors, stream);

    TSNE::get_distances(X, n, p, indices, distances, n_neighbors, stream);


    // Normalize distances
    if (verbose) printf("[Info] Now normalizing distances so exp(D) doesn't explode.\n");
    TSNE::normalize_distances(n, distances, n * n_neighbors, stream);


    // Optimal perplexity
    if (verbose) printf("[Info] Searching for optimal perplexity via bisection search.\n");
    float *P = (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);

    // Determine best blocksize / gridsize
    int blockSize_N = 1024; // default to 1024
    int minGridSize_N;
    hipOccupancyMaxPotentialBlockSize(&minGridSize_N, &blockSize_N, __determine_sigmas, 0, n);
    const int gridSize_N = ceildiv(n, blockSize_N);


    const float P_sum = TSNE::determine_sigmas(distances, P, perplexity, perplexity_max_iter,
                                        perplexity_tol, n, n_neighbors, stream, gridSize_N, blockSize_N, handle);
    d_alloc->deallocate(distances, sizeof(float) * n * n_neighbors, stream);
    if (verbose) printf("[Info] Perplexity sum = %f\n", P_sum);


    // Convert data to COO layout
    MLCommon::Sparse::COO<float> P_PT;
    TSNE::symmetrize_perplexity(P, indices, &P_PT, n, n_neighbors, P_sum, early_exaggeration, stream, handle);
        
    const int NNZ = P_PT.nnz;
    float *VAL = P_PT.vals;
    const int *COL = P_PT.rows;
    const int *ROW = P_PT.cols;


    // Allocate data [NOTICE Fortran Contiguous for method = Naive and C-Contiguous for fast]
    if (initialize_embeddings)
        TSNE::random_vector(Y, -0.03f, 0.03f, n * n_components, stream, seed);


    // Allocate space
    if (verbose) printf("[Info] Now allocating memory for TSNE.\n");
    float *norm = (float *)d_alloc->allocate(sizeof(float) * n, stream);
    float *Q_sum = (float *)d_alloc->allocate(sizeof(float) * n, stream);
    double *sum = (double *)d_alloc->allocate(sizeof(double), stream);

    float *attract = (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);
    float *repel = (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);

    float *iY = (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);
    float *gains = (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);
    float *means = (float*)d_alloc->allocate(sizeof(float) * n_components, stream);


    // Compute optimal gridSize and blockSize for attractive forces
    int blockSize_NNZ = 1024; // default to 1024
    int minGridSize_NNZ;
    if (n_components == 2)
        hipOccupancyMaxPotentialBlockSize(&minGridSize_NNZ, &blockSize_NNZ, __attractive_fast_2dim, 0, NNZ);
    else
        hipOccupancyMaxPotentialBlockSize(&minGridSize_NNZ, &blockSize_NNZ, __attractive_fast, 0, NNZ);
    const int gridSize_NNZ = ceildiv(NNZ, blockSize_NNZ);

    // Compute optimal gridSize and blockSize for applying forces
    int blockSize_dimN = 1024; // default to 1024
    int minGridSize_dimN;
    hipOccupancyMaxPotentialBlockSize(&minGridSize_dimN, &blockSize_dimN, __apply_forces, 0, n*n_components);
    const int gridSize_dimN = ceildiv(n*n_components, blockSize_dimN);


    // Do gradient updates
    float momentum = pre_momentum;
    float Z;

    if (verbose) printf("[Info] Start gradient updates!\n");
    for (int iter = 0; iter < max_iter; iter++) {
        if (iter == exaggeration_iter) {
            momentum = post_momentum;
            // Divide perplexities
            const float div = 1.0f / early_exaggeration;
            array_multiply(VAL, NNZ, div, stream);
        }
        // Get norm(Y)
        get_norm_fast(Y, norm, n, k, stream, gridSize_N, blockSize_N);
        
        // Fast compute attractive forces from COO matrix
        attractive_fast(VAL, COL, ROW, Y, norm, attract, NNZ, n, n_components, stream,
            gridSize_NNZ, blockSize_NNZ);

        // Fast compute repulsive forces
        Z = repulsive_fast(Y, repel, norm, Q_sum, n, n_components, stream);
        if (verbose && iter % 100 == 0)
            printf("[Info]  Z at iter = %d is %lf.\n", iter, Z);

        // Integrate forces with momentum
        apply_forces(attract, means, repel, Y, iY, gains, n, k, Z, min_gain, momentum, eta, stream,
            gridSize_dimN, blockSize_dimN);
    }

    printf("[Info]  TSNE has finished!\n");
    // Clean up
    P_PT.destroy();

    d_alloc->deallocate(norm, sizeof(float) * n, stream);
    d_alloc->deallocate(Q_sum, sizeof(float) * n, stream);
    d_alloc->deallocate(sum, sizeof(double), stream);

    d_alloc->deallocate(attract, sizeof(float) * n * k, stream);
    d_alloc->deallocate(repel, sizeof(float) * n * k, stream);

    d_alloc->deallocate(iY, sizeof(float) * n * k, stream);
    d_alloc->deallocate(gains, sizeof(float) * n * k, stream);
    d_alloc->deallocate(means, sizeof(float) * k, stream);
}


}  // namespace ML
