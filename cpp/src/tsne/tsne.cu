#include "hip/hip_runtime.h"

#pragma once

#include "common/cumlHandle.hpp"

#include "tsne/tsne.h"

#include "hipblas.h"
#include "distances.h"
#include "kernels.h"
#include "utils.h"

namespace ML {
using namespace MLCommon;

void TSNE(const cumlHandle &handle, const float *X, float *Y, const int n,
          const int p, const int n_components, const int n_neighbors,
          const float perplexity, const int perplexity_epochs,
          const int perplexity_tol, const float early_exaggeration,
          const int exaggeration_iter, const float min_gain, const float eta,
          const int epochs, const float pre_momentum, const float post_momentum,
          const long long seed) {
  auto d_alloc = handle.getDeviceAllocator();

  hipStream_t stream = handle.getStream();

  // Some preliminary intializations for cuBLAS and cuML
  DEBUG("[Info]	Create cuBLAS and cuML handles.\n");
  const int k = n_components;

  hipblasHandle_t blas_handle = handle.getImpl().getCublasHandle();

  const float neg2 = -2.0f, beta = 0.0f, one = 1.0f;

  random_vector(Y, -0.05f, 0.05f, n * k, seed, false, stream);

  // Get distances
  DEBUG("[Info]	Get distances\n");
  float *distances =
    (float *)d_alloc->allocate(n * n_neighbors * sizeof(float), stream);
  long *indices =
    (long *)d_alloc->allocate(sizeof(long) * n * n_neighbors, stream);
  get_distances(X, n, p, indices, distances, n_neighbors, stream);

  // Get perplexity
  DEBUG("[Info]	Get perplexity\n");
  float *P = (float *)d_alloc->allocate(
    sizeof(float) * n * n_neighbors,
    stream);  //cmalloc(sizeof(float)*n*n_neighbors, false);
  float P_sum = determine_sigmas(distances, P, perplexity, perplexity_epochs,
                                 perplexity_tol, n, n_neighbors);
  d_alloc->deallocate(distances, n * n_neighbors * sizeof(float), stream);
  DEBUG("[Info]	P_sum = %f\n", P_sum);

  // Convert data to COO layout
  DEBUG("[Info]	Convert to COO layout and symmetrize\n");
  COO_t<float> P_PT;
  symmetrize_perplexity(P, indices, &P_PT, n, n_neighbors, P_sum,
                        early_exaggeration, stream);

  // Allocate data [NOTICE all Fortran Contiguous]
  DEBUG("[Info]	Malloc data and space\n");
  float *noise = (float *)d_alloc->allocate(sizeof(float) * n * n, stream);

  float *Q = (float *)d_alloc->allocate(sizeof(float) * n * n, stream);
  float *norm = (float *)d_alloc->allocate(sizeof(float) * n, stream);
  float *Q_sum = (float *)d_alloc->allocate(sizeof(float) * n, stream);
  float *sum = (float *)d_alloc->allocate(sizeof(float), stream);

  float *attract = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);
  float *repel = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);

  float *iY = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);
  float *gains = (float *)d_alloc->allocate(sizeof(float) * n * k, stream);

  // Do gradient updates
  float momentum = pre_momentum;
  float Z;
  int error;

  DEBUG("[Info]	Start iterations\n");
  for (int iter = 0; iter < epochs; iter++) {
    if (iter == 100) momentum = post_momentum;

    if (iter == exaggeration_iter) {
      float div = 1.0f / early_exaggeration;
      inplace_multiply(P_PT.vals, P_PT.nnz, div);
    }

    // Get norm(Y)
    get_norm(Y, norm, n, k);
#if IF_DEBUG
    printf("[Info]	Norm(y)\n\n");
    std::cout << MLCommon::arr2Str(norm, 20, "norm", stream) << std::endl;
#endif

    // Do -2 * (Y @ Y.T)
    if (error = hipblasSsyrk(blas_handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n,
                            k, &neg2, Y, n, &beta, Q, n)) {
      DEBUG("[ERROR]	Error from BLAS = %d", error);
      break;
    }
#if IF_DEBUG
    printf("[Info]	Y @ Y.T\n\n");
    std::cout << MLCommon::arr2Str(Q, 20, "YYT", stream) << std::endl;
#endif

    // Form T = 1 / (1+d)
    Z = form_t_distribution(Q, norm, n, Q_sum, sum);
    DEBUG("[Info]	Z =  %lf iter = %d\n", Z, iter);
#if IF_DEBUG
    printf("[Info]	Q 1/(1+d)\n\n");
    std::cout << MLCommon::arr2Str(Q, 20, "QQ", stream);

    printf("[Info]	Q_sum\n\n");
    std::cout << MLCommon::arr2Str(Q_sum, 20, "Q_sum", stream);

    // float sum_;
    // hipMemcpy(&sum_, sum, sizeof(float), hipMemcpyDeviceToHost);
    // printf("[Info]	sum again = %lf\n\n", sum_);
#endif

    // Compute attractive forces with COO matrix
    attractive_forces(P_PT.vals, P_PT.cols, P_PT.rows, Q, Y, attract, P_PT.nnz,
                      n, k);

    // Change Q to Q**2 for repulsion
    postprocess_Q(Q, Q_sum, n);

    // Compute repel_1 = Q @ Y
    if (error =
          hipblasSsymm(blas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, n,
                      k, &one, Q, n, Y, n, &beta, repel, n)) {
      DEBUG("[ERROR]	Error from BLAS = %d", error);
      break;
    }

    // Repel_2 = mean contributions yi - yj
    // Repel = Repel_1 - Repel_2
    repel_minus_QY(repel, Q_sum, Y, n, k);

    // Integrate forces with momentum
    apply_forces(attract, repel, Y, iY, noise, gains, n, k, Z, min_gain,
                 momentum, eta);

#if IF_DEBUG
    break;
#endif
  }

  P_PT.destroy();

  d_alloc->deallocate(noise, sizeof(float) * n * n, stream);

  d_alloc->deallocate(Q, sizeof(float) * n * n, stream);
  d_alloc->deallocate(norm, sizeof(float) * n, stream);
  d_alloc->deallocate(Q_sum, sizeof(float) * n, stream);
  d_alloc->deallocate(sum, sizeof(float), stream);

  d_alloc->deallocate(attract, sizeof(float) * n * k, stream);
  d_alloc->deallocate(repel, sizeof(float) * n * k, stream);

  d_alloc->deallocate(iY, sizeof(float) * n * k, stream);
  d_alloc->deallocate(gains, sizeof(float) * n * k, stream);
}

}  // namespace ML
