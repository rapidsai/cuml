#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "../../src_prims/utils.h"
#include "distances.h"
#include "exact_kernels.h"
#include "tsne/tsne.h"
#include "utils.h"

#include "barnes_hut.h"
#include "exact_tsne.h"

namespace ML {

/**
 * @brief Dimensionality reduction via TSNE using either Barnes Hut O(NlogN) or brute force O(N^2).
 * @input param handle: The GPU handle.
 * @input param X: The dataset you want to apply TSNE on.
 * @output param Y: The final embedding. Will overwrite this internally.
 * @input param n: Number of rows in data X.
 * @input param p: Number of columns in data X.
 * @input param dim: Number of output dimensions for embeddings Y.
 * @input param n_neighbors: Number of nearest neighbors used.
 * @input param theta: Float between 0 and 1. Tradeoff for speed (0) vs accuracy (1) for Barnes Hut only.
 * @input param epssq: A tiny jitter to promote numerical stability.
 * @input param perplexity: How many nearest neighbors are used during the construction of Pij.
 * @input param perplexity_max_iter: Number of iterations used to construct Pij.
 * @input param perplexity_tol: The small tolerance used for Pij to ensure numerical stability.
 * @input param early_exaggeration: How much early pressure you want the clusters in TSNE to spread out more.
 * @input param exaggeration_iter: How many iterations you want the early pressure to run for.
 * @input param min_gain: Rounds up small gradient updates.
 * @input param pre_learning_rate: The learning rate during the exaggeration phase.
 * @input param post_learning_rate: The learning rate after the exaggeration phase.
 * @input param max_iter: The maximum number of iterations TSNE should run for.
 * @input param min_grad_norm: The smallest gradient norm TSNE should terminate on.
 * @input param pre_momentum: The momentum used during the exaggeration phase.
 * @input param post_momentum: The momentum used after the exaggeration phase.
 * @input param random_state: Set this to -1 for pure random intializations or >= 0 for reproducible outputs.
 * @input param verbose: Whether to print error messages or not.
 * @input param intialize_embeddings: Whether to overwrite the current Y vector with random noise.
 * @input param barnes_hut: Whether to use the fast Barnes Hut or use the slower exact version.
 */
void TSNE_fit(const cumlHandle &handle, const float *X, float *Y, const int n,
              const int p, const int dim, int n_neighbors, const float theta,
              const float epssq, float perplexity,
              const int perplexity_max_iter, const float perplexity_tol,
              const float early_exaggeration, const int exaggeration_iter,
              const float min_gain, const float pre_learning_rate,
              const float post_learning_rate, const int max_iter,
              const float min_grad_norm, const float pre_momentum,
              const float post_momentum, const long long random_state,
              const bool verbose, const bool intialize_embeddings,
              bool barnes_hut) {
  ASSERT(n > 0 && p > 0 && dim > 0 && n_neighbors > 0 && X != NULL && Y != NULL,
         "Wrong input args");
  if (dim > 2 and barnes_hut) {
    barnes_hut = false;
    printf(
      "[Warn]  Barnes Hut only works for dim == 2. Switching to exact "
      "solution.\n");
  }
  if (n_neighbors > n) n_neighbors = n;
  if (n_neighbors > 1023) {
    printf("[Warn]  FAISS only supports maximum n_neighbors = 1023.\n");
    n_neighbors = 1023;
  }
  // Perplexity must be less than number of datapoints
  // "How to Use t-SNE Effectively" https://distill.pub/2016/misread-tsne/
  if (perplexity > n) perplexity = n;

  if (verbose) {
    printf("[Info]  Data size = (%d, %d) with dim = %d perplexity = %f\n", n, p,
           dim, perplexity);
    if (perplexity < 5 or perplexity > 50)
      printf(
        "[Warn]  Perplexity should be within ranges (5, 50). Your results "
        "might be a bit strange...\n");
    if (n_neighbors < perplexity * 3.0f)
      printf(
        "[Warn]  # of Nearest Neighbors should be at least 3 * perplexity. "
        "Your results might be a bit strange...\n");
  }

  auto d_alloc = handle.getDeviceAllocator();
  hipStream_t stream = handle.getStream();

  START_TIMER;
  //---------------------------------------------------
  // Get distances
  if (verbose) printf("[Info] Getting distances.\n");
  float *distances =
    (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
  long *indices =
    (long *)d_alloc->allocate(sizeof(long) * n * n_neighbors, stream);
  TSNE::get_distances(X, n, p, indices, distances, n_neighbors, stream);
  //---------------------------------------------------
  END_TIMER(DistancesTime);

  START_TIMER;
  //---------------------------------------------------
  // Normalize distances
  if (verbose)
    printf("[Info] Now normalizing distances so exp(D) doesn't explode.\n");
  TSNE::normalize_distances(n, distances, n_neighbors, stream);
  //---------------------------------------------------
  END_TIMER(NormalizeTime);

  START_TIMER;
  //---------------------------------------------------
  // Optimal perplexity
  if (verbose)
    printf("[Info] Searching for optimal perplexity via bisection search.\n");
  float *P =
    (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
  const float P_sum =
    TSNE::perplexity_search(distances, P, perplexity, perplexity_max_iter,
                            perplexity_tol, n, n_neighbors, handle);
  d_alloc->deallocate(distances, sizeof(float) * n * n_neighbors, stream);
  if (verbose) printf("[Info] Perplexity sum = %f\n", P_sum);
  //---------------------------------------------------
  END_TIMER(PerplexityTime);

  START_TIMER;
  //---------------------------------------------------
  // Convert data to COO layout
  MLCommon::Sparse::COO<float> COO_Matrix;
  TSNE::symmetrize_perplexity(P, indices, n, n_neighbors, P_sum,
                              early_exaggeration, &COO_Matrix, stream, handle);
  const int NNZ = COO_Matrix.nnz;
  float *VAL = COO_Matrix.vals;
  const int *COL = COO_Matrix.cols;
  const int *ROW = COO_Matrix.rows;
  //---------------------------------------------------
  END_TIMER(SymmetrizeTime);

  if (barnes_hut) {
    TSNE::Barnes_Hut(VAL, COL, ROW, NNZ, handle, Y, n, theta, epssq,
                     early_exaggeration, exaggeration_iter, min_gain,
                     pre_learning_rate, post_learning_rate, max_iter,
                     min_grad_norm, pre_momentum, post_momentum, random_state,
                     verbose);
  } else {
    TSNE::Exact_TSNE(VAL, COL, ROW, NNZ, handle, Y, n, dim, early_exaggeration,
                     exaggeration_iter, min_gain, pre_learning_rate,
                     post_learning_rate, max_iter, min_grad_norm, pre_momentum,
                     post_momentum, random_state, verbose,
                     intialize_embeddings);
  }

  COO_Matrix.destroy();
}

}  // namespace ML
