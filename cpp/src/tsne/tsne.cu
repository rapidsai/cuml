#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#pragma once

#include "common/cumlHandle.hpp"

#include "distances.h"
#include "fast_kernels.h"
#include "linalg/norm.h"
#include "tsne/tsne.h"
#include "utils.h"

namespace ML {
using MLCommon::ceildiv;

void TSNE_fit(const cumlHandle &handle, const float *X, float *Y, const int n,
              const int p, const int n_components, int n_neighbors,

              float perplexity, const int perplexity_max_iter,
              const int perplexity_tol,

              const float early_exaggeration, const int exaggeration_iter,

              const float min_gain, const float gains_add,
              const float gains_mult, const float eta, const int max_iter,
              const float pre_momentum, const float post_momentum,

              const long long seed, const bool initialize_embeddings,
              const bool verbose)
// Method = 0 for Naive, 1 for Fast
{
  assert(n > 0 && p > 0 && n_components > 0 && n_neighbors > 0 && X != NULL &&
         Y != NULL);
  auto d_alloc = handle.getDeviceAllocator();
  hipStream_t stream = handle.getStream();

  if (n_neighbors > n) {
    printf(
      "[Warn]  Notice n = %d, n_neighbors = %d. n_neighbors must be <= n!\n", n,
      n_neighbors);
    n_neighbors = n;
  }
  if (perplexity >= n) {
    printf("[Warn]  Notice n = %d, perplexity = %f. perplexity must be <= n!\n",
           n, perplexity);
    perplexity = n;
  }
  if (verbose)
    printf(
      "[Info]  Data = (%d, %d) with n_components = %d and perplexity = %f\n", n,
      p, n_components, perplexity);

  // Get distances
  if (verbose) printf("[Info] Getting distances.\n");

  float *distances =
    (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);
  long *indices =
    (long *)d_alloc->allocate(sizeof(long) * n * n_neighbors, stream);

  TSNE::get_distances(X, n, p, indices, distances, n_neighbors, stream);

  // Normalize distances
  if (verbose)
    printf("[Info] Now normalizing distances so exp(D) doesn't explode.\n");
  TSNE::normalize_distances(n, distances, n * n_neighbors, stream);

  // Optimal perplexity
  if (verbose)
    printf("[Info] Searching for optimal perplexity via bisection search.\n");
  float *P =
    (float *)d_alloc->allocate(sizeof(float) * n * n_neighbors, stream);

  // Determine best blocksize / gridsize
  int blockSize_N = 1024;  // default to 1024
  int minGridSize_N;
  hipOccupancyMaxPotentialBlockSize(&minGridSize_N, &blockSize_N,
                                     TSNE::__determine_sigmas, 0, n);
  const int gridSize_N = ceildiv(n, blockSize_N);

  const float P_sum = TSNE::determine_sigmas(
    distances, P, perplexity, perplexity_max_iter, perplexity_tol, n,
    n_neighbors, stream, gridSize_N, blockSize_N, handle);
  d_alloc->deallocate(distances, sizeof(float) * n * n_neighbors, stream);
  if (verbose) printf("[Info] Perplexity sum = %f\n", P_sum);

  // Convert data to COO layout
  MLCommon::Sparse::COO<float> P_PT;
  TSNE::symmetrize_perplexity(P, indices, &P_PT, n, n_neighbors, P_sum,
                              early_exaggeration, stream, handle);

  const int NNZ = P_PT.nnz;
  float *VAL = P_PT.vals;
  const int *COL = P_PT.rows;
  const int *ROW = P_PT.cols;

  // Allocate data [NOTICE Fortran Contiguous for method = Naive and C-Contiguous for fast]
  if (initialize_embeddings)
    TSNE::random_vector(Y, -0.03f, 0.03f, n * n_components, stream, seed);

  // Allocate space
  if (verbose) printf("[Info] Now allocating memory for TSNE.\n");
  float *norm = (float *)d_alloc->allocate(sizeof(float) * n, stream);
  float *Q_sum = (float *)d_alloc->allocate(sizeof(float) * n, stream);
  double *sum = (double *)d_alloc->allocate(sizeof(double), stream);

  float *attract =
    (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);
  float *repel =
    (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);

  float *iY =
    (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);
  float *gains =
    (float *)d_alloc->allocate(sizeof(float) * n * n_components, stream);
  float *means =
    (float *)d_alloc->allocate(sizeof(float) * n_components, stream);

  // Compute optimal gridSize and blockSize for attractive forces
  int blockSize_NNZ = 1024;  // default to 1024
  int minGridSize_NNZ;
  if (n_components == 2)
    hipOccupancyMaxPotentialBlockSize(&minGridSize_NNZ, &blockSize_NNZ,
                                       TSNE::__attractive_fast_2dim, 0, NNZ);
  else
    hipOccupancyMaxPotentialBlockSize(&minGridSize_NNZ, &blockSize_NNZ,
                                       TSNE::__attractive_fast, 0, NNZ);
  const int gridSize_NNZ = ceildiv(NNZ, blockSize_NNZ);

  // Compute optimal gridSize and blockSize for applying forces
  int blockSize_dimN = 1024;  // default to 1024
  int minGridSize_dimN;
  hipOccupancyMaxPotentialBlockSize(&minGridSize_dimN, &blockSize_dimN,
                                     TSNE::__apply_forces, 0, n * n_components);
  const int gridSize_dimN = ceildiv(n * n_components, blockSize_dimN);

  // Do gradient updates
  float momentum = pre_momentum;
  float Z;

  if (verbose) printf("[Info] Start gradient updates!\n");
  for (int iter = 0; iter < max_iter; iter++) {
    if (iter == exaggeration_iter) {
      momentum = post_momentum;
      // Divide perplexities
      const float div = 1.0f / early_exaggeration;
      MLCommon::LinAlg::scalarMultiply(VAL, (const float *)VAL, div, NNZ,
                                       stream);
    }
    // Get norm(Y)
    MLCommon::LinAlg::rowNorm(norm, Y, n_components, n,
                              MLCommon::LinAlg::L2Norm, false, stream);
    //TSNE::get_norm_fast(Y, norm, n, n_components, stream, gridSize_N, blockSize_N);

    // Fast compute attractive forces from COO matrix
    TSNE::attractive_fast(VAL, COL, ROW, Y, norm, attract, NNZ, n, n_components,
                          stream, gridSize_NNZ, blockSize_NNZ);

    // Fast compute repulsive forces
    Z = TSNE::repulsive_fast(Y, repel, norm, Q_sum, n, n_components, stream);
    if (verbose && iter % 100 == 0)
      printf("[Info]  Z at iter = %d is %lf.\n", iter, Z);

    // Integrate forces with momentum
    TSNE::apply_forces(attract, means, repel, Y, iY, gains, n, n_components, Z,
                       min_gain, momentum, eta, stream, gridSize_dimN,
                       blockSize_dimN, gains_add, gains_mult);

    if (momentum > 0.8) momentum -= 0.005;
  }

  printf("[Info]  TSNE has finished!\n");
  // Clean up
  P_PT.destroy();

  d_alloc->deallocate(norm, sizeof(float) * n, stream);
  d_alloc->deallocate(Q_sum, sizeof(float) * n, stream);
  d_alloc->deallocate(sum, sizeof(double), stream);

  d_alloc->deallocate(attract, sizeof(float) * n * n_components, stream);
  d_alloc->deallocate(repel, sizeof(float) * n * n_components, stream);

  d_alloc->deallocate(iY, sizeof(float) * n * n_components, stream);
  d_alloc->deallocate(gains, sizeof(float) * n * n_components, stream);
  d_alloc->deallocate(means, sizeof(float) * n_components, stream);
}

}  // namespace ML
