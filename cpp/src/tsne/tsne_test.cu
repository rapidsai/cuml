/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef IF_DEBUG
#define IF_DEBUG 1
#endif

#include <gtest/gtest.h>
#include <score/scores.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include "tsne/digits.h"
#include "tsne/tsne.cu"

#include "cuda_utils.h"

using namespace MLCommon;
using namespace MLCommon::Score;
using namespace MLCommon::Distance;

using namespace ML;

class TSNETest : public ::testing::Test {
 protected:
  void basicTest() {
    cumlHandle handle;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    float *X_d, *Y_d;
    MLCommon::allocate(X_d, n * p);
    MLCommon::allocate(Y_d, n * 2);
    MLCommon::updateDevice(X_d, digits.data(), n * p, stream);

    std::cout << "[>>>>]    Starting TSNE....\n";
    TSNE_fit(handle, X_d, Y_d, n, p, 2, 90);
    std::cout << "[>>>>]    Got embeddings!....\n";

    std::cout << "Updating host" << std::endl;
    float *embeddings_h = (float *)malloc(sizeof(float) * n * 2);
    hipMemcpy(embeddings_h, Y_d, sizeof(float) * n * 2,
               hipMemcpyDeviceToHost);

    int k = 0;
    float C_contiguous_embedding[n * 2];
    for (int i = 0; i < n; i++) {
      for (int j = 0; j < 2; j++)
        C_contiguous_embedding[k++] = embeddings_h[j * n + i];
    }

    float *YY;
    MLCommon::allocate(YY, n * 2);
    MLCommon::updateDevice(YY, C_contiguous_embedding, n * 2, stream);

    std::cout << "DONE!" << std::endl;

    CUDA_CHECK(hipPeekAtLastError());

    // Test trustworthiness
    // euclidean test
    score = trustworthiness_score<float, EucUnexpandedL2>(
      X_d, YY, n, p, 2, 5, handle.getDeviceAllocator(), stream);

    std::cout << "SCORE: " << score << std::endl;

    free(embeddings_h);
    CUDA_CHECK(hipFree(Y_d));
    CUDA_CHECK(hipFree(YY));
    CUDA_CHECK(hipFree(X_d));

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {}

 protected:
  int n = 1797;
  int p = 64;
  double score;
};

typedef TSNETest TSNETestF;
TEST_F(TSNETestF, Result) { ASSERT_TRUE(0.98 < score); }