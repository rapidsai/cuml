/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/common/logger.hpp>
#include <cuml/neighbors/knn_sparse.hpp>

#include <raft/sparse/selection/knn.cuh>

#include <hipsparse.h>

namespace ML {
namespace Sparse {

void brute_force_knn(raft::handle_t &handle, const int *idx_indptr,
                     const int *idx_indices, const float *idx_data,
                     size_t idx_nnz, int n_idx_rows, int n_idx_cols,
                     const int *query_indptr, const int *query_indices,
                     const float *query_data, size_t query_nnz,
                     int n_query_rows, int n_query_cols, int *output_indices,
                     float *output_dists, int k,
                     size_t batch_size_index,  // approx 1M
                     size_t batch_size_query,
                     raft::distance::DistanceType metric, float metricArg) {
  auto d_alloc = handle.get_device_allocator();
  hipsparseHandle_t cusparse_handle = handle.get_cusparse_handle();
  hipStream_t stream = handle.get_stream();

  raft::sparse::selection::brute_force_knn(
    idx_indptr, idx_indices, idx_data, idx_nnz, n_idx_rows, n_idx_cols,
    query_indptr, query_indices, query_data, query_nnz, n_query_rows,
    n_query_cols, output_indices, output_dists, k, cusparse_handle, d_alloc,
    stream, batch_size_index, batch_size_query, metric, metricArg);
}
};  // namespace Sparse
};  // namespace ML
