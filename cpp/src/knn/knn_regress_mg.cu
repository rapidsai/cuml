/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "knn_opg_common.cuh"

namespace ML {
namespace KNN {
namespace opg {

using namespace knn_common;

template struct KNN_RE_params<float, int64_t, float, float>;

void knn_regress(raft::handle_t &handle,
                 std::vector<Matrix::Data<float> *> *out,
                 std::vector<Matrix::floatData_t *> &idx_data,
                 Matrix::PartDescriptor &idx_desc,
                 std::vector<Matrix::floatData_t *> &query_data,
                 Matrix::PartDescriptor &query_desc,
                 std::vector<std::vector<float *>> &y, bool rowMajorIndex,
                 bool rowMajorQuery, int k, int n_outputs, size_t batch_size,
                 bool verbose) {
  KNN_RE_params<float, int64_t, float, float> params(
    knn_operation::regression, &idx_data, &idx_desc, &query_data, &query_desc,
    rowMajorIndex, rowMajorQuery, k, batch_size, verbose, n_outputs, &y, out);

  cuda_utils cutils(handle);
  opg_knn(params, cutils);
}
};  // namespace opg
};  // namespace KNN
};  // namespace ML
