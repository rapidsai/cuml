/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cumlHandle.hpp>

#include <cuml/common/logger.hpp>
#include <cuml/neighbors/knn.hpp>

#include <ml_mg_utils.cuh>

#include <label/classlabels.cuh>
#include <selection/knn.cuh>

#include <hip/hip_runtime.h>
#include <cuda_utils.cuh>

#include <sstream>
#include <vector>

namespace ML {

void brute_force_knn(raft::handle_t &handle, std::vector<float *> &input,
                     std::vector<int> &sizes, int D, float *search_items, int n,
                     int64_t *res_I, float *res_D, int k, bool rowMajorIndex,
                     bool rowMajorQuery, MetricType metric, float metric_arg,
                     bool expanded) {
  ASSERT(input.size() == sizes.size(),
         "input and sizes vectors must be the same size");

  std::vector<hipStream_t> int_streams = handle.get_internal_streams();

  MLCommon::Selection::brute_force_knn(
    input, sizes, D, search_items, n, res_I, res_D, k,
    handle.get_device_allocator(), handle.get_stream(), int_streams.data(),
    handle.get_num_internal_streams(), rowMajorIndex, rowMajorQuery, nullptr,
    metric, metric_arg, expanded);
}

void knn_classify(raft::handle_t &handle, int *out, int64_t *knn_indices,
                  std::vector<int *> &y, size_t n_index_rows,
                  size_t n_query_rows, int k) {
  auto d_alloc = handle.get_device_allocator();
  hipStream_t stream = handle.get_stream();

  std::vector<int *> uniq_labels(y.size());
  std::vector<int> n_unique(y.size());

  for (int i = 0; i < y.size(); i++) {
    MLCommon::Label::getUniqueLabels(y[i], n_index_rows, &(uniq_labels[i]),
                                     &(n_unique[i]), stream, d_alloc);
  }

  MLCommon::Selection::knn_classify(out, knn_indices, y, n_index_rows,
                                    n_query_rows, k, uniq_labels, n_unique,
                                    d_alloc, stream);
}

void knn_regress(raft::handle_t &handle, float *out, int64_t *knn_indices,
                 std::vector<float *> &y, size_t n_index_rows,
                 size_t n_query_rows, int k) {
  MLCommon::Selection::knn_regress(out, knn_indices, y, n_index_rows,
                                   n_query_rows, k, handle.get_stream());
}

void knn_class_proba(raft::handle_t &handle, std::vector<float *> &out,
                     int64_t *knn_indices, std::vector<int *> &y,
                     size_t n_index_rows, size_t n_query_rows, int k) {
  auto d_alloc = handle.get_device_allocator();
  hipStream_t stream = handle.get_stream();

  std::vector<int *> uniq_labels(y.size());
  std::vector<int> n_unique(y.size());

  for (int i = 0; i < y.size(); i++) {
    MLCommon::Label::getUniqueLabels(y[i], n_index_rows, &(uniq_labels[i]),
                                     &(n_unique[i]), stream, d_alloc);
  }

  MLCommon::Selection::class_probs(out, knn_indices, y, n_index_rows,
                                   n_query_rows, k, uniq_labels, n_unique,
                                   d_alloc, stream);
}

/**
 * @brief Flat C API function to perform a brute force knn on
 * a series of input arrays and combine the results into a single
 * output array for indexes and distances.
 *
 * @param[in] handle the cuml handle to use
 * @param[in] input an array of pointers to the input arrays
 * @param[in] sizes an array of sizes of input arrays
 * @param[in] n_params array size of input and sizes
 * @param[in] D the dimensionality of the arrays
 * @param[in] search_items array of items to search of dimensionality D
 * @param[in] n number of rows in search_items
 * @param[out] res_I the resulting index array of size n * k
 * @param[out] res_D the resulting distance array of size n * k
 * @param[in] k the number of nearest neighbors to return
 * @param[in] rowMajorIndex is the index array in row major layout?
 * @param[in] rowMajorQuery is the query array in row major layout?
 */
extern "C" cumlError_t knn_search(const cumlHandle_t handle, float **input,
                                  int *sizes, int n_params, int D,
                                  float *search_items, int n, int64_t *res_I,
                                  float *res_D, int k, bool rowMajorIndex,
                                  bool rowMajorQuery, int metric_type,
                                  float metric_arg, bool expanded) {
  cumlError_t status;

  raft::handle_t *handle_ptr;
  std::tie(handle_ptr, status) = ML::handleMap.lookupHandlePointer(handle);

  std::vector<hipStream_t> int_streams = handle_ptr->get_internal_streams();

  std::vector<float *> input_vec(n_params);
  std::vector<int> sizes_vec(n_params);
  for (int i = 0; i < n_params; i++) {
    input_vec.push_back(input[i]);
    sizes_vec.push_back(sizes[i]);
  }

  if (status == CUML_SUCCESS) {
    try {
      MLCommon::Selection::brute_force_knn(
        input_vec, sizes_vec, D, search_items, n, res_I, res_D, k,
        handle_ptr->get_device_allocator(), handle_ptr->get_stream(),
        int_streams.data(), handle_ptr->get_num_internal_streams(),
        rowMajorIndex, rowMajorQuery, nullptr, (ML::MetricType)metric_type,
        metric_arg, expanded);
    } catch (...) {
      status = CUML_ERROR_UNKNOWN;
    }
  }
  return status;
}
};  // END NAMESPACE ML
