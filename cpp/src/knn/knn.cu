/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common/cumlHandle.hpp"

#include <cuml/common/logger.hpp>
#include <cuml/neighbors/knn.hpp>

#include "ml_mg_utils.h"

#include "label/classlabels.h"
#include "selection/knn.h"

#include <hip/hip_runtime.h>
#include "cuda_utils.h"

#include <sstream>
#include <vector>

namespace ML {

void brute_force_knn(cumlHandle &handle, std::vector<float *> &input,
                     std::vector<int> &sizes, int D, float *search_items, int n,
                     int64_t *res_I, float *res_D, int k, bool rowMajorIndex,
                     bool rowMajorQuery) {
  ASSERT(input.size() == sizes.size(),
         "input and sizes vectors must be the same size");

  std::vector<hipStream_t> int_streams = handle.getImpl().getInternalStreams();

  MLCommon::Selection::brute_force_knn(
    input, sizes, D, search_items, n, res_I, res_D, k,
    handle.getImpl().getDeviceAllocator(), handle.getImpl().getStream(),
    int_streams.data(), handle.getImpl().getNumInternalStreams(), rowMajorIndex,
    rowMajorQuery);
}

void knn_classify(cumlHandle &handle, int *out, int64_t *knn_indices,
                  std::vector<int *> &y, size_t n_labels, size_t n_samples,
                  int k) {
  auto d_alloc = handle.getDeviceAllocator();
  hipStream_t stream = handle.getStream();

  std::vector<int *> uniq_labels(y.size());
  std::vector<int> n_unique(y.size());

  for (int i = 0; i < y.size(); i++) {
    MLCommon::Label::getUniqueLabels(y[i], n_samples, &(uniq_labels[i]),
                                     &(n_unique[i]), stream, d_alloc);
  }

  MLCommon::Selection::knn_classify(out, knn_indices, y, n_labels, n_samples, k,
                                    uniq_labels, n_unique, d_alloc, stream);
}

void knn_regress(cumlHandle &handle, float *out, int64_t *knn_indices,
                 std::vector<float *> &y, size_t n_labels, size_t n_samples,
                 int k) {
  MLCommon::Selection::knn_regress(out, knn_indices, y, n_labels, n_samples, k,
                                   handle.getStream());
}

void knn_class_proba(cumlHandle &handle, std::vector<float *> &out,
                     int64_t *knn_indices, std::vector<int *> &y,
                     size_t n_labels, size_t n_samples, int k) {
  auto d_alloc = handle.getDeviceAllocator();
  hipStream_t stream = handle.getStream();

  std::vector<int *> uniq_labels(y.size());
  std::vector<int> n_unique(y.size());

  for (int i = 0; i < y.size(); i++) {
    MLCommon::Label::getUniqueLabels(y[i], n_samples, &(uniq_labels[i]),
                                     &(n_unique[i]), stream, d_alloc);
  }

  MLCommon::Selection::class_probs(out, knn_indices, y, n_labels, n_samples, k,
                                   uniq_labels, n_unique, d_alloc, stream);
}

kNN::kNN(const cumlHandle &handle, int D, int verbosity)
  : D(D), total_n(0), indices(0) {
  ML::Logger::get().setLevel(verbosity);
  this->handle = const_cast<cumlHandle *>(&handle);
  sizes = nullptr;
  ptrs = nullptr;
}

kNN::~kNN() {
  if (this->indices > 0) {
    reset();
  }
}

void kNN::reset() {
  if (this->indices > 0) {
    this->indices = 0;
    this->total_n = 0;

    delete[] this->ptrs;
    delete[] this->sizes;
  }
}

/**
	 * Fit a kNN model by creating separate indices for multiple given
	 * instances of kNNParams.
	 * @param input  an array of pointers to data on (possibly different) devices
	 * @param N 	 number of items in input array.
	 * @param rowMajor is the input in rowMajor?
	 */
void kNN::fit(std::vector<float *> &input, std::vector<int> &sizes,
              bool rowMajor) {
  this->rowMajorIndex = rowMajor;

  int N = input.size();

  CUML_LOG_DEBUG("N=%d", N);

  reset();

  this->indices = N;
  this->ptrs = (float **)malloc(N * sizeof(float *));
  this->sizes = (int *)malloc(N * sizeof(int));

  for (int i = 0; i < N; i++) {
    this->ptrs[i] = input[i];
    this->sizes[i] = sizes[i];
  }
}

/**
	 * Search the kNN for the k-nearest neighbors of a set of query vectors
	 * @param search_items set of vectors to query for neighbors
	 * @param n 		   number of items in search_items
	 * @param res_I 	   pointer to device memory for returning k nearest indices
	 * @param res_D		   pointer to device memory for returning k nearest distances
	 * @param k			   number of neighbors to query
	 * @param rowMajor is the query array in row major layout?
	 */
void kNN::search(float *search_items, int n, int64_t *res_I, float *res_D,
                 int k, bool rowMajor) {
  ASSERT(this->indices > 0, "Cannot search before model has been trained.");

  std::vector<hipStream_t> int_streams =
    handle->getImpl().getInternalStreams();

  MLCommon::Selection::brute_force_knn(
    ptrs, sizes, indices, D, search_items, n, res_I, res_D, k,
    handle->getImpl().getDeviceAllocator(), handle->getImpl().getStream(),
    int_streams.data(), handle->getImpl().getNumInternalStreams(),
    this->rowMajorIndex, rowMajor);
}
};  // namespace ML

/**
 * @brief Flat C API function to perform a brute force knn on
 * a series of input arrays and combine the results into a single
 * output array for indexes and distances.
 *
 * @param handle the cuml handle to use
 * @param input an array of pointers to the input arrays
 * @param sizes an array of sizes of input arrays
 * @param n_params array size of input and sizes
 * @param D the dimensionality of the arrays
 * @param search_items array of items to search of dimensionality D
 * @param n number of rows in search_items
 * @param res_I the resulting index array of size n * k
 * @param res_D the resulting distance array of size n * k
 * @param k the number of nearest neighbors to return
 * @param rowMajorIndex is the index array in row major layout?
 * @param rowMajorQuery is the query array in row major layout?
 */
extern "C" cumlError_t knn_search(const cumlHandle_t handle, float **input,
                                  int *sizes, int n_params, int D,
                                  float *search_items, int n, int64_t *res_I,
                                  float *res_D, int k, bool rowMajorIndex,
                                  bool rowMajorQuery) {
  cumlError_t status;

  ML::cumlHandle *handle_ptr;
  std::tie(handle_ptr, status) = ML::handleMap.lookupHandlePointer(handle);

  std::vector<hipStream_t> int_streams =
    handle_ptr->getImpl().getInternalStreams();

  std::vector<float *> input_vec(n_params);
  std::vector<int> sizes_vec(n_params);
  for (int i = 0; i < n_params; i++) {
    input_vec.push_back(input[i]);
    sizes_vec.push_back(sizes[i]);
  }

  if (status == CUML_SUCCESS) {
    try {
      MLCommon::Selection::brute_force_knn(
        input_vec, sizes_vec, D, search_items, n, res_I, res_D, k,
        handle_ptr->getImpl().getDeviceAllocator(),
        handle_ptr->getImpl().getStream(), int_streams.data(),
        handle_ptr->getImpl().getNumInternalStreams(), rowMajorIndex,
        rowMajorQuery);
    } catch (...) {
      status = CUML_ERROR_UNKNOWN;
    }
  }
  return status;
}
