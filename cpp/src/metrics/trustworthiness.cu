/*
 * Copyright (c) 2018-2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda_utils.h>
#include <common/cumlHandle.hpp>
#include "distance/distance.h"
#include "score/scores.h"

namespace ML {
namespace Metrics {

/**
        * @brief Compute the trustworthiness score
        * @input param X: Data in original dimension
        * @input param X_embedded: Data in target dimension (embedding)
        * @input param n: Number of samples
        * @input param m: Number of features in high/original dimension
        * @input param d: Number of features in low/embedded dimension
        * @input param n_neighbors: Number of neighbors considered by trustworthiness score
        * @input tparam distance_type: Distance type to consider
        * @return Trustworthiness score
        */
template <typename math_t, MLCommon::Distance::DistanceType distance_type>
double trustworthiness_score(const cumlHandle& h, math_t* X, math_t* X_embedded,
                             int n, int m, int d, int n_neighbors,
                             int batchSize) {
  hipStream_t stream = h.getStream();
  auto d_alloc = h.getDeviceAllocator();

  return MLCommon::Score::trustworthiness_score<math_t, distance_type>(
    X, X_embedded, n, m, d, n_neighbors, d_alloc, stream, batchSize);
}

template double
trustworthiness_score<float, MLCommon::Distance::EucUnexpandedL2Sqrt>(
  const cumlHandle& h, float* X, float* X_embedded, int n, int m, int d,
  int n_neighbors, int batchSize);

};  //end namespace Metrics
};  //end namespace ML
