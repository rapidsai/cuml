
/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/metrics/metrics.hpp>

#include <raft/core/handle.hpp>

#include <cuvs/distance/distance.hpp>
#include <cuvs/stats/silhouette_score.hpp>

namespace ML {

namespace Metrics {
double silhouette_score(const raft::handle_t& handle,
                        double* y,
                        int nRows,
                        int nCols,
                        int* labels,
                        int nLabels,
                        double* silScores,
                        raft::distance::DistanceType metric)
{
  std::optional<raft::device_vector_view<double, int64_t>> silhouette_score_per_sample;
  if (silScores != NULL) {
    silhouette_score_per_sample = raft::make_device_vector_view<double, int64_t>(silScores, nRows);
  }

  return cuvs::stats::silhouette_score(
    handle,
    raft::make_device_matrix_view<const double, int64_t>(y, nRows, nCols),
    raft::make_device_vector_view<const int, int64_t>(labels, nRows),
    silhouette_score_per_sample,
    nLabels,
    static_cast<cuvs::distance::DistanceType>(metric));
}
}  // namespace Metrics
}  // namespace ML
