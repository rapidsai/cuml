#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *  http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifdef _OPENMP
#include <omp.h>
#else
#define omp_get_max_threads() 1
#endif
#include <treelite/c_api.h>
#include <treelite/tree.h>
#include <cstdio>
#include <cuml/ensemble/randomforest.hpp>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#include "randomforest_impl.cuh"

namespace ML {

using namespace MLCommon;
using namespace std;
namespace tl = treelite;

/**
 * @brief Set RF_metrics.
 * @param[in] rf_type: Random Forest type: classification or regression
 * @param[in] cfg_accuracy: accuracy.
 * @param[in] mean_abs_error: mean absolute error.
 * @param[in] mean_squared_error: mean squared error.
 * @param[in] median_abs_error: median absolute error.
 * @return RF_metrics struct with classification or regression score.
 */
RF_metrics set_all_rf_metrics(RF_type rf_type, float accuracy,
                              double mean_abs_error, double mean_squared_error,
                              double median_abs_error) {
  RF_metrics rf_metrics;
  rf_metrics.rf_type = rf_type;
  rf_metrics.accuracy = accuracy;
  rf_metrics.mean_abs_error = mean_abs_error;
  rf_metrics.mean_squared_error = mean_squared_error;
  rf_metrics.median_abs_error = median_abs_error;
  return rf_metrics;
}

/**
 * @brief Set RF_metrics for classification.
 * @param[in] cfg_accuracy: accuracy.
 * @return RF_metrics struct with classification score.
 */
RF_metrics set_rf_metrics_classification(float accuracy) {
  return set_all_rf_metrics(RF_type::CLASSIFICATION, accuracy, -1.0, -1.0,
                            -1.0);
}

/**
 * @brief Set RF_metrics for regression.
 * @param[in] mean_abs_error: mean absolute error.
 * @param[in] mean_squared_error: mean squared error.
 * @param[in] median_abs_error: median absolute error.
 * @return RF_metrics struct with regression score.
 */
RF_metrics set_rf_metrics_regression(double mean_abs_error,
                                     double mean_squared_error,
                                     double median_abs_error) {
  return set_all_rf_metrics(RF_type::REGRESSION, -1.0, mean_abs_error,
                            mean_squared_error, median_abs_error);
}

/**
 * @brief Print either accuracy metric for classification, or mean absolute error,
 *   mean squared error, and median absolute error metrics for regression.
 * @param[in] rf_metrics: random forest metrics to print.
 */
void print(const RF_metrics rf_metrics) {
  if (rf_metrics.rf_type == RF_type::CLASSIFICATION) {
    std::cout << "Accuracy: " << rf_metrics.accuracy << std::endl;
  } else if (rf_metrics.rf_type == RF_type::REGRESSION) {
    std::cout << "Mean Absolute Error: " << rf_metrics.mean_abs_error
              << std::endl;
    std::cout << "Mean Squared Error: " << rf_metrics.mean_squared_error
              << std::endl;
    std::cout << "Median Absolute Error: " << rf_metrics.median_abs_error
              << std::endl;
  }
}

/**
 * @brief Update labels so they are unique from 0 to n_unique_labels values.
 *   Create/update an old label to new label map per random forest.
 * @param[in] n_rows: number of rows (labels)
 * @param[in,out] labels: 1D labels array to be changed in-place.
 * @param[in,out] labels_map: map of old label values to new ones.
 * @param[in] verbose: debugging flag.
 */
void preprocess_labels(int n_rows, std::vector<int>& labels,
                       std::map<int, int>& labels_map, bool verbose) {
  std::pair<std::map<int, int>::iterator, bool> ret;
  int n_unique_labels = 0;

  if (verbose) std::cout << "Preprocessing labels\n";
  for (int i = 0; i < n_rows; i++) {
    ret = labels_map.insert(std::pair<int, int>(labels[i], n_unique_labels));
    if (ret.second) {
      n_unique_labels += 1;
    }
    if (verbose) std::cout << "Mapping " << labels[i] << " to ";
    labels[i] = ret.first->second;  //Update labels **IN-PLACE**
    if (verbose) std::cout << labels[i] << std::endl;
  }
  if (verbose) std::cout << "Finished preprocessing labels\n";
}

/**
 * @brief Revert label preprocessing effect, if needed.
 * @param[in] n_rows: number of rows (labels)
 * @param[in,out] labels: 1D labels array to be changed in-place.
 * @param[in] labels_map: map of old to new label values used during preprocessing.
 * @param[in] verbose: debugging flag.
 */
void postprocess_labels(int n_rows, std::vector<int>& labels,
                        std::map<int, int>& labels_map, bool verbose) {
  if (verbose) std::cout << "Postrocessing labels\n";
  std::map<int, int>::iterator it;
  int n_unique_cnt = labels_map.size();
  std::vector<int> reverse_map;
  reverse_map.resize(n_unique_cnt);
  for (auto it = labels_map.begin(); it != labels_map.end(); it++) {
    reverse_map[it->second] = it->first;
  }

  for (int i = 0; i < n_rows; i++) {
    if (verbose)
      std::cout << "Mapping " << labels[i] << " back to "
                << reverse_map[labels[i]] << std::endl;
    labels[i] = reverse_map[labels[i]];
  }
  if (verbose) std::cout << "Finished postrocessing labels\n";
}

/**
 * @brief Set RF_params parameters members; use default tree parameters.
 * @param[in,out] params: update with random forest parameters
 * @param[in] cfg_n_trees: number of trees; default 1
 * @param[in] cfg_bootstrap: bootstrapping; default true
 * @param[in] cfg_rows_sample: rows sample; default 1.0f
 * @param[in] cfg_n_streams: No of parallel CUDA for training forest
 */
void set_rf_params(RF_params& params, int cfg_n_trees, bool cfg_bootstrap,
                   float cfg_rows_sample, int cfg_seed, int cfg_n_streams) {
  params.n_trees = cfg_n_trees;
  params.bootstrap = cfg_bootstrap;
  params.rows_sample = cfg_rows_sample;
  params.seed = cfg_seed;
  params.n_streams = min(cfg_n_streams, omp_get_max_threads());
  if (params.n_streams == cfg_n_streams) {
    std::cout << "Warning! Max setting Max streams to max openmp threads "
              << omp_get_max_threads() << std::endl;
  }
  if (cfg_n_trees < params.n_streams) params.n_streams = cfg_n_trees;
  set_tree_params(params.tree_params);  // use default tree params
}

/**
 * @brief Set all RF_params parameters members, including tree parameters.
 * @param[in,out] params: update with random forest parameters
 * @param[in] cfg_n_trees: number of trees
 * @param[in] cfg_bootstrap: bootstrapping
 * @param[in] cfg_rows_sample: rows sample
 * @param[in] cfg_n_streams: No of parallel CUDA for training forest
 * @param[in] cfg_tree_params: tree parameters
 */
void set_all_rf_params(RF_params& params, int cfg_n_trees, bool cfg_bootstrap,
                       float cfg_rows_sample, int cfg_seed, int cfg_n_streams,
                       DecisionTree::DecisionTreeParams cfg_tree_params) {
  params.n_trees = cfg_n_trees;
  params.bootstrap = cfg_bootstrap;
  params.rows_sample = cfg_rows_sample;
  params.seed = cfg_seed;
  params.n_streams = min(cfg_n_streams, omp_get_max_threads());
  if (cfg_n_trees < params.n_streams) params.n_streams = cfg_n_trees;
  set_tree_params(params.tree_params);  // use input tree params
  params.tree_params = cfg_tree_params;
}

/**
 * @brief Check validity of all random forest hyper-parameters.
 * @param[in] rf_params: random forest hyper-parameters
 */
void validity_check(const RF_params rf_params) {
  ASSERT((rf_params.n_trees > 0), "Invalid n_trees %d", rf_params.n_trees);
  ASSERT((rf_params.rows_sample > 0) && (rf_params.rows_sample <= 1.0),
         "rows_sample value %f outside permitted (0, 1] range",
         rf_params.rows_sample);
  DecisionTree::validity_check(rf_params.tree_params);
}

/**
 * @brief Print all random forest hyper-parameters.
 * @param[in] rf_params: random forest hyper-parameters
 */
void print(const RF_params rf_params) {
  std::cout << "n_trees: " << rf_params.n_trees << std::endl;
  std::cout << "bootstrap: " << rf_params.bootstrap << std::endl;
  std::cout << "rows_sample: " << rf_params.rows_sample << std::endl;
  std::cout << "n_streams: " << rf_params.n_streams << std::endl;
  DecisionTree::print(rf_params.tree_params);
}

/**
 * @brief Set the trees pointer of RandomForestMetaData to nullptr.
 * @param[in, out] forest: CPU pointer to RandomForestMetaData.
 */
template <class T, class L>
void null_trees_ptr(RandomForestMetaData<T, L>*& forest) {
  forest->trees = nullptr;
}

/**
 * @brief Print summary for all trees in the random forest.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 * @param[in] forest: CPU pointer to RandomForestMetaData struct.
 */
template <class T, class L>
void print_rf_summary(const RandomForestMetaData<T, L>* forest) {
  if (!forest || !forest->trees) {
    std::cout << "Empty forest" << std::endl;
  } else {
    std::cout << "Forest has " << forest->rf_params.n_trees
              << " trees, max_depth "
              << forest->rf_params.tree_params.max_depth;
    std::cout << ", and max_leaves " << forest->rf_params.tree_params.max_leaves
              << std::endl;
    for (int i = 0; i < forest->rf_params.n_trees; i++) {
      std::cout << "Tree #" << i << std::endl;
      DecisionTree::print_tree_summary<T, L>(&(forest->trees[i]));
    }
  }
}

/**
 * @brief Print detailed view of all trees in the random forest.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 * @param[in] forest: CPU pointer to RandomForestMetaData struct.
 */
template <class T, class L>
void print_rf_detailed(const RandomForestMetaData<T, L>* forest) {
  if (!forest || !forest->trees) {
    std::cout << "Empty forest" << std::endl;
  } else {
    std::cout << "Forest has " << forest->rf_params.n_trees
              << " trees, max_depth "
              << forest->rf_params.tree_params.max_depth;
    std::cout << ", and max_leaves " << forest->rf_params.tree_params.max_leaves
              << std::endl;
    for (int i = 0; i < forest->rf_params.n_trees; i++) {
      std::cout << "Tree #" << i << std::endl;
      DecisionTree::print_tree<T, L>(&(forest->trees[i]));
    }
  }
}

template <class T, class L>
void build_treelite_forest(ModelHandle* model,
                           const RandomForestMetaData<T, L>* forest,
                           int num_features, int task_category,
                           std::vector<unsigned char>& data) {
  bool check_val = (data).empty();
  if (not check_val) {
    // create a temp file
    const char* filename = std::tmpnam(nullptr);
    // write the model bytes into the temp file
    std::ofstream file(filename, std::ios::binary);
    file.write((char*)&data[0], data.size());
    // read the file as a protobuf model
    TREELITE_CHECK(TreeliteLoadProtobufModel(filename, model));
  }

  else {
    // Non-zero value here for random forest models.
    // The value should be set to 0 if the model is gradient boosted trees.
    int random_forest_flag = 1;
    ModelBuilderHandle model_builder;
    // num_output_group is 1 for binary classification and regression
    // num_output_group is #class for multiclass classification which is the same as task_category
    int num_output_group = task_category > 2 ? task_category : 1;
    TREELITE_CHECK(TreeliteCreateModelBuilder(
      num_features, num_output_group, random_forest_flag, &model_builder));

    if (task_category > 2) {
      // Multi-class classification
      TREELITE_CHECK(TreeliteModelBuilderSetModelParam(
        model_builder, "pred_transform", "max_index"));
    }

    for (int i = 0; i < forest->rf_params.n_trees; i++) {
      DecisionTree::TreeMetaDataNode<T, L>* tree_ptr = &forest->trees[i];
      TreeBuilderHandle tree_builder;

      TREELITE_CHECK(TreeliteCreateTreeBuilder(&tree_builder));
      if (tree_ptr->sparsetree.size() != 0) {
        DecisionTree::build_treelite_tree<T, L>(tree_builder, tree_ptr,
                                                num_output_group);

        // The third argument -1 means append to the end of the tree list.
        TREELITE_CHECK(
          TreeliteModelBuilderInsertTree(model_builder, tree_builder, -1));
      }
    }

    TREELITE_CHECK(TreeliteModelBuilderCommitModel(model_builder, model));
    TREELITE_CHECK(TreeliteDeleteModelBuilder(model_builder));
  }
}

std::vector<unsigned char> save_model(ModelHandle model) {
  // create a temp file
  const char* filename = std::tmpnam(nullptr);
  // export the treelite model to protobuf nd save it in the temp file
  TreeliteExportProtobufModel(filename, model);
  // read from the temp file and obtain the model bytes
  std::ifstream in(filename, std::ifstream::ate | std::ifstream::binary);
  in.seekg(0, std::ios::end);
  int size_of_file = in.tellg();
  vector<unsigned char> bytes_info(size_of_file, 0);
  ifstream infile(filename, ios::in | ios::binary);
  infile.read((char*)&bytes_info[0], bytes_info.size());
  return bytes_info;
}

/**
 * @brief Compares the trees present in concatenated treelite forest with the trees
 *   of the forests present in the different workers. If there is a difference in the two
 *   then an error statement will be thrown.
 * @param[in] tree_from_concatenated_forest: Tree info from the concatenated forest.
 * @param[in] tree_from_individual_forest: Tree info from the forest present in each worker.
 */
void compare_trees(tl::Tree& tree_from_concatenated_forest,
                   tl::Tree& tree_from_individual_forest) {
  ASSERT(tree_from_concatenated_forest.num_nodes ==
           tree_from_individual_forest.num_nodes,
         "Error! Mismatch the number of nodes present in a tree in the "
         "concatenated forest and"
         " the tree present in the individual forests");
  for (int each_node = 0; each_node < tree_from_concatenated_forest.num_nodes;
       each_node++) {
    tl::Tree::Node& node_from_concat = tree_from_concatenated_forest[each_node];
    tl::Tree::Node& node_from_indiv = tree_from_individual_forest[each_node];
    ASSERT(node_from_concat.is_root() == node_from_indiv.is_root(),
           "Error! root position mismatch between concatenated forest and the"
           " individual forests ");
    ASSERT(node_from_concat.parent() == node_from_indiv.parent(),
           "Error! node parent mismatch between concatenated forest and the"
           " individual forests ");
    ASSERT(node_from_concat.is_leaf() == node_from_indiv.is_leaf(),
           "Error! mismatch in the position of a leaf between concatenated "
           "forest and the"
           " individual forests ");
    ASSERT(node_from_concat.leaf_value() == node_from_indiv.leaf_value(),
           "Error! leaf value mismatch between concatenated forest and the"
           " individual forests ");
    ASSERT(node_from_concat.cright() == node_from_indiv.cright(),
           "Error! mismatch in the position of the node between concatenated "
           "forest and the"
           " individual forests ");
    ASSERT(node_from_concat.cleft() == node_from_indiv.cleft(),
           "Error! mismatch in the position of the node between concatenated "
           "forest and the"
           " individual forests ");
    ASSERT(
      node_from_concat.split_index() == node_from_indiv.split_index(),
      "Error! split index value mismatch between concatenated forest and the"
      " individual forests ");
  }
}

/**
 * @brief Compares the concatenated treelite model with the information of the forest
 *   present in the different workers. If there is a difference in the two then an error
 *   statement will be thrown.
 * @param[in] concat_tree_handle: ModelHandle for the concatenated forest.
 * @param[in] treelite_handles: List containing ModelHandles for the forest present in
 *   each worker.
 */
void compare_concat_forest_to_subforests(
  ModelHandle concat_tree_handle, std::vector<ModelHandle> treelite_handles) {
  size_t concat_forest;
  size_t total_num_trees = 0;
  for (int forest_idx = 0; forest_idx < treelite_handles.size(); forest_idx++) {
    size_t num_trees_each_forest;
    TREELITE_CHECK(TreeliteQueryNumTree(treelite_handles[forest_idx],
                                        &num_trees_each_forest));
    total_num_trees = total_num_trees + num_trees_each_forest;
  }

  TREELITE_CHECK(TreeliteQueryNumTree(concat_tree_handle, &concat_forest));

  ASSERT(
    concat_forest == total_num_trees,
    "Error! the number of trees in the concatenated forest and the sum "
    "of the trees present in the forests present in each worker are not equal");

  int concat_mod_tree_num = 0;
  tl::Model& concat_model = *(tl::Model*)(concat_tree_handle);
  for (int forest_idx = 0; forest_idx < treelite_handles.size(); forest_idx++) {
    tl::Model& model = *(tl::Model*)(treelite_handles[forest_idx]);

    ASSERT(
      concat_model.num_feature == model.num_feature,
      "Error! number of features mismatch between concatenated forest and the"
      " individual forests ");
    ASSERT(concat_model.num_output_group == model.num_output_group,
           "Error! number of output group mismatch between concatenated forest "
           "and the"
           " individual forests ");
    ASSERT(concat_model.random_forest_flag == model.random_forest_flag,
           "Error! random forest flag value mismatch between concatenated "
           "forest and the"
           " individual forests ");

    for (int indiv_trees = 0; indiv_trees < model.trees.size(); indiv_trees++) {
      compare_trees(concat_model.trees[concat_mod_tree_num + indiv_trees],
                    model.trees[indiv_trees]);
    }
    concat_mod_tree_num = concat_mod_tree_num + model.trees.size();
  }
}

/**
 * @brief Concatenates the forest information present in different workers to
 *  create a single forest. This concatenated forest is stored in a new treelite model.
 *  The model created is owned by and must be freed by the user.
 * @param[in] concat_tree_handle: ModelHandle for the concatenated forest.
 * @param[in] treelite_handles: List containing ModelHandles for the forest present in
 *   each worker.
 */
ModelHandle concatenate_trees(std::vector<ModelHandle> treelite_handles) {
  tl::Model& first_model = *(tl::Model*)treelite_handles[0];
  tl::Model* concat_model = new tl::Model;
  for (int forest_idx = 0; forest_idx < treelite_handles.size(); forest_idx++) {
    tl::Model& model = *(tl::Model*)treelite_handles[forest_idx];
    concat_model->trees.insert(concat_model->trees.end(), model.trees.begin(),
                               model.trees.end());
  }
  concat_model->num_feature = first_model.num_feature;
  concat_model->num_output_group = first_model.num_output_group;
  concat_model->random_forest_flag = first_model.random_forest_flag;
  concat_model->param = first_model.param;
  return concat_model;
}

/**
 * @defgroup Random Forest Classification - Fit function
 * @brief Build (i.e., fit, train) random forest classifier for input data.
 * @param[in] user_handle: cumlHandle
 * @param[in,out] forest: CPU pointer to RandomForestMetaData object. User allocated.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format,
 *   excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (int only), with one label per
 *   training sample. Device pointer.
 *   Assumption: labels were preprocessed to map to ascending numbers from 0;
 *   needed for current gini impl. in decision tree
 * @param[in] n_unique_labels: #unique label values (known during preprocessing)
 * @param[in] rf_params: Random Forest training hyper parameter struct.
 * @{
 */
void fit(const cumlHandle& user_handle, RandomForestClassifierF*& forest,
         float* input, int n_rows, int n_cols, int* labels, int n_unique_labels,
         RF_params rf_params) {
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<float, int>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfClassifier<float>> rf_classifier =
    std::make_shared<rfClassifier<float>>(rf_params);
  rf_classifier->fit(user_handle, input, n_rows, n_cols, labels,
                     n_unique_labels, forest);
}

void fit(const cumlHandle& user_handle, RandomForestClassifierD*& forest,
         double* input, int n_rows, int n_cols, int* labels,
         int n_unique_labels, RF_params rf_params) {
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<double, int>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfClassifier<double>> rf_classifier =
    std::make_shared<rfClassifier<double>>(rf_params);
  rf_classifier->fit(user_handle, input, n_rows, n_cols, labels,
                     n_unique_labels, forest);
}
/** @} */

/**
 * @defgroup Random Forest Classification - Predict function
 * @brief Predict target feature for input data; n-ary classification for
     single feature supported.
 * @param[in] user_handle: cumlHandle.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 * @{
 */
void predict(const cumlHandle& user_handle,
             const RandomForestClassifierF* forest, const float* input,
             int n_rows, int n_cols, int* predictions, bool verbose) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<float>> rf_classifier =
    std::make_shared<rfClassifier<float>>(forest->rf_params);
  rf_classifier->predict(user_handle, input, n_rows, n_cols, predictions,
                         forest, verbose);
}

void predict(const cumlHandle& user_handle,
             const RandomForestClassifierD* forest, const double* input,
             int n_rows, int n_cols, int* predictions, bool verbose) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<double>> rf_classifier =
    std::make_shared<rfClassifier<double>>(forest->rf_params);
  rf_classifier->predict(user_handle, input, n_rows, n_cols, predictions,
                         forest, verbose);
}
/** @} */

/**
 * @defgroup Random Forest Classification - Predict function
 * @brief Predict target feature for input data; n-ary classification for
     single feature supported.
 * @param[in] user_handle: cumlHandle.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 * @{
 */
void predictGetAll(const cumlHandle& user_handle,
                   const RandomForestClassifierF* forest, const float* input,
                   int n_rows, int n_cols, int* predictions, bool verbose) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<float>> rf_classifier =
    std::make_shared<rfClassifier<float>>(forest->rf_params);
  rf_classifier->predictGetAll(user_handle, input, n_rows, n_cols, predictions,
                               forest, verbose);
}

void predictGetAll(const cumlHandle& user_handle,
                   const RandomForestClassifierD* forest, const double* input,
                   int n_rows, int n_cols, int* predictions, bool verbose) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<double>> rf_classifier =
    std::make_shared<rfClassifier<double>>(forest->rf_params);
  rf_classifier->predictGetAll(user_handle, input, n_rows, n_cols, predictions,
                               forest, verbose);
}
/** @} */

/**
 * @defgroup Random Forest Classification - Score function
 * @brief Compare predicted features validate against ref_labels.
 * @param[in] user_handle: cumlHandle.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 * @return RF_metrics struct with classification score (i.e., accuracy)
 * @{
 */
RF_metrics score(const cumlHandle& user_handle,
                 const RandomForestClassifierF* forest, const int* ref_labels,
                 int n_rows, const int* predictions, bool verbose) {
  RF_metrics classification_score = rfClassifier<float>::score(
    user_handle, ref_labels, n_rows, predictions, verbose);
  return classification_score;
}

RF_metrics score(const cumlHandle& user_handle,
                 const RandomForestClassifierD* forest, const int* ref_labels,
                 int n_rows, const int* predictions, bool verbose) {
  RF_metrics classification_score = rfClassifier<double>::score(
    user_handle, ref_labels, n_rows, predictions, verbose);
  return classification_score;
}

RF_params set_rf_class_obj(int max_depth, int max_leaves, float max_features,
                           int n_bins, int split_algo, int min_rows_per_node,
                           float min_impurity_decrease, bool bootstrap_features,
                           bool bootstrap, int n_trees, float rows_sample,
                           int seed, CRITERION split_criterion,
                           bool quantile_per_tree, int cfg_n_streams) {
  DecisionTree::DecisionTreeParams tree_params;
  DecisionTree::set_tree_params(
    tree_params, max_depth, max_leaves, max_features, n_bins, split_algo,
    min_rows_per_node, min_impurity_decrease, bootstrap_features,
    split_criterion, quantile_per_tree);
  RF_params rf_params;
  set_all_rf_params(rf_params, n_trees, bootstrap, rows_sample, seed,
                    cfg_n_streams, tree_params);
  return rf_params;
}

/** @} */

/**
 * @defgroup Random Forest Regression - Fit function
 * @brief Build (i.e., fit, train) random forest regressor for input data.
 * @param[in] user_handle: cumlHandle
 * @param[in,out] forest: CPU pointer to RandomForestMetaData object. User allocated.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format,
 *   excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (float or double), with one label per
 *   training sample. Device pointer.
 * @param[in] rf_params: Random Forest training hyper parameter struct.
 * @{
 */
void fit(const cumlHandle& user_handle, RandomForestRegressorF*& forest,
         float* input, int n_rows, int n_cols, float* labels,
         RF_params rf_params) {
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<float, float>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfRegressor<float>> rf_regressor =
    std::make_shared<rfRegressor<float>>(rf_params);
  rf_regressor->fit(user_handle, input, n_rows, n_cols, labels, forest);
}

void fit(const cumlHandle& user_handle, RandomForestRegressorD*& forest,
         double* input, int n_rows, int n_cols, double* labels,
         RF_params rf_params) {
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<double, double>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfRegressor<double>> rf_regressor =
    std::make_shared<rfRegressor<double>>(rf_params);
  rf_regressor->fit(user_handle, input, n_rows, n_cols, labels, forest);
}
/** @} */

/**
 * @defgroup Random Forest Regression - Predict function
 * @brief Predict target feature for input data; regression for single feature supported.
 * @param[in] user_handle: cumlHandle.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 * @{
 */
void predict(const cumlHandle& user_handle,
             const RandomForestRegressorF* forest, const float* input,
             int n_rows, int n_cols, float* predictions, bool verbose) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfRegressor<float>> rf_regressor =
    std::make_shared<rfRegressor<float>>(forest->rf_params);
  rf_regressor->predict(user_handle, input, n_rows, n_cols, predictions, forest,
                        verbose);
}

void predict(const cumlHandle& user_handle,
             const RandomForestRegressorD* forest, const double* input,
             int n_rows, int n_cols, double* predictions, bool verbose) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfRegressor<double>> rf_regressor =
    std::make_shared<rfRegressor<double>>(forest->rf_params);
  rf_regressor->predict(user_handle, input, n_rows, n_cols, predictions, forest,
                        verbose);
}
/** @} */

/**
 * @defgroup Random Forest Regression - Score function
 * @brief Predict target feature for input data and validate against ref_labels.
 * @param[in] user_handle: cumlHandle.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 * @return RF_metrics struct with regression score (i.e., mean absolute error,
 *   mean squared error, median absolute error)
 * @{
 */
RF_metrics score(const cumlHandle& user_handle,
                 const RandomForestRegressorF* forest, const float* ref_labels,
                 int n_rows, const float* predictions, bool verbose) {
  RF_metrics regression_score = rfRegressor<float>::score(
    user_handle, ref_labels, n_rows, predictions, verbose);

  return regression_score;
}

RF_metrics score(const cumlHandle& user_handle,
                 const RandomForestRegressorD* forest, const double* ref_labels,
                 int n_rows, const double* predictions, bool verbose) {
  RF_metrics regression_score = rfRegressor<double>::score(
    user_handle, ref_labels, n_rows, predictions, verbose);
  return regression_score;
}
/** @} */

// Functions' specializations
template void print_rf_summary<float, int>(
  const RandomForestClassifierF* forest);
template void print_rf_summary<double, int>(
  const RandomForestClassifierD* forest);
template void print_rf_summary<float, float>(
  const RandomForestRegressorF* forest);
template void print_rf_summary<double, double>(
  const RandomForestRegressorD* forest);

template void print_rf_detailed<float, int>(
  const RandomForestClassifierF* forest);
template void print_rf_detailed<double, int>(
  const RandomForestClassifierD* forest);
template void print_rf_detailed<float, float>(
  const RandomForestRegressorF* forest);
template void print_rf_detailed<double, double>(
  const RandomForestRegressorD* forest);

template void null_trees_ptr<float, int>(RandomForestClassifierF*& forest);
template void null_trees_ptr<double, int>(RandomForestClassifierD*& forest);
template void null_trees_ptr<float, float>(RandomForestRegressorF*& forest);
template void null_trees_ptr<double, double>(RandomForestRegressorD*& forest);

template void build_treelite_forest<float, int>(
  ModelHandle* model, const RandomForestMetaData<float, int>* forest,
  int num_features, int task_category, std::vector<unsigned char>& data);
template void build_treelite_forest<double, int>(
  ModelHandle* model, const RandomForestMetaData<double, int>* forest,
  int num_features, int task_category, std::vector<unsigned char>& data);
template void build_treelite_forest<float, float>(
  ModelHandle* model, const RandomForestMetaData<float, float>* forest,
  int num_features, int task_category, std::vector<unsigned char>& data);
template void build_treelite_forest<double, double>(
  ModelHandle* model, const RandomForestMetaData<double, double>* forest,
  int num_features, int task_category, std::vector<unsigned char>& data);
}  // End namespace ML