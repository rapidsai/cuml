#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *  http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifdef _OPENMP
#include <omp.h>
#else
#define omp_get_max_threads() 1
#endif

#include <cuml/ensemble/randomforest.hpp>

#include <cuml/tree/flatnode.h>
#include <treelite/c_api.h>
#include <treelite/tree.h>

#include <cuml/common/logger.hpp>
#include <raft/error.hpp>

#include "randomforest_impl.cuh"

#include <cstdio>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

namespace ML {

using namespace MLCommon;
using namespace std;
namespace tl = treelite;

/**
 * @brief Set RF_metrics.
 * @param[in] rf_type: Random Forest type: classification or regression
 * @param[in] cfg_accuracy: accuracy.
 * @param[in] mean_abs_error: mean absolute error.
 * @param[in] mean_squared_error: mean squared error.
 * @param[in] median_abs_error: median absolute error.
 * @return RF_metrics struct with classification or regression score.
 */
RF_metrics set_all_rf_metrics(RF_type rf_type, float accuracy,
                              double mean_abs_error, double mean_squared_error,
                              double median_abs_error) {
  RF_metrics rf_metrics;
  rf_metrics.rf_type = rf_type;
  rf_metrics.accuracy = accuracy;
  rf_metrics.mean_abs_error = mean_abs_error;
  rf_metrics.mean_squared_error = mean_squared_error;
  rf_metrics.median_abs_error = median_abs_error;
  return rf_metrics;
}

/**
 * @brief Set RF_metrics for classification.
 * @param[in] cfg_accuracy: accuracy.
 * @return RF_metrics struct with classification score.
 */
RF_metrics set_rf_metrics_classification(float accuracy) {
  return set_all_rf_metrics(RF_type::CLASSIFICATION, accuracy, -1.0, -1.0,
                            -1.0);
}

/**
 * @brief Set RF_metrics for regression.
 * @param[in] mean_abs_error: mean absolute error.
 * @param[in] mean_squared_error: mean squared error.
 * @param[in] median_abs_error: median absolute error.
 * @return RF_metrics struct with regression score.
 */
RF_metrics set_rf_metrics_regression(double mean_abs_error,
                                     double mean_squared_error,
                                     double median_abs_error) {
  return set_all_rf_metrics(RF_type::REGRESSION, -1.0, mean_abs_error,
                            mean_squared_error, median_abs_error);
}

/**
 * @brief Print either accuracy metric for classification, or mean absolute error,
 *   mean squared error, and median absolute error metrics for regression.
 * @param[in] rf_metrics: random forest metrics to print.
 */
void print(const RF_metrics rf_metrics) {
  if (rf_metrics.rf_type == RF_type::CLASSIFICATION) {
    CUML_LOG_DEBUG("Accuracy: %f", rf_metrics.accuracy);
  } else if (rf_metrics.rf_type == RF_type::REGRESSION) {
    CUML_LOG_DEBUG("Mean Absolute Error: %f", rf_metrics.mean_abs_error);
    CUML_LOG_DEBUG("Mean Squared Error: %f", rf_metrics.mean_squared_error);
    CUML_LOG_DEBUG("Median Absolute Error: %f", rf_metrics.median_abs_error);
  }
}

/**
 * @brief Update labels so they are unique from 0 to n_unique_labels values.
 *   Create/update an old label to new label map per random forest.
 * @param[in] n_rows: number of rows (labels)
 * @param[in,out] labels: 1D labels array to be changed in-place.
 * @param[in,out] labels_map: map of old label values to new ones.
 * @param[in] verbosity: verbosity level for logging messages during execution
 */
void preprocess_labels(int n_rows, std::vector<int>& labels,
                       std::map<int, int>& labels_map, int verbosity) {
  std::pair<std::map<int, int>::iterator, bool> ret;
  int n_unique_labels = 0;
  ML::Logger::get().setLevel(verbosity);

  CUML_LOG_DEBUG("Preprocessing labels");
  for (int i = 0; i < n_rows; i++) {
    ret = labels_map.insert(std::pair<int, int>(labels[i], n_unique_labels));
    if (ret.second) {
      n_unique_labels += 1;
    }
    auto prev = labels[i];
    labels[i] = ret.first->second;  //Update labels **IN-PLACE**
    CUML_LOG_DEBUG("Mapping %d to %d", prev, labels[i]);
  }
  CUML_LOG_DEBUG("Finished preprocessing labels");
}

/**
 * @brief Revert label preprocessing effect, if needed.
 * @param[in] n_rows: number of rows (labels)
 * @param[in,out] labels: 1D labels array to be changed in-place.
 * @param[in] labels_map: map of old to new label values used during preprocessing.
 * @param[in] verbosity: verbosity level for logging messages during execution
 */
void postprocess_labels(int n_rows, std::vector<int>& labels,
                        std::map<int, int>& labels_map, int verbosity) {
  ML::Logger::get().setLevel(verbosity);
  CUML_LOG_DEBUG("Postrocessing labels");
  std::map<int, int>::iterator it;
  int n_unique_cnt = labels_map.size();
  std::vector<int> reverse_map;
  reverse_map.resize(n_unique_cnt);
  for (auto it = labels_map.begin(); it != labels_map.end(); it++) {
    reverse_map[it->second] = it->first;
  }

  for (int i = 0; i < n_rows; i++) {
    auto prev = labels[i];
    labels[i] = reverse_map[prev];
    CUML_LOG_DEBUG("Mapping %d back to %d", prev, labels[i]);
  }
  CUML_LOG_DEBUG("Finished postrocessing labels");
}

/**
 * @brief Check validity of all random forest hyper-parameters.
 * @param[in] rf_params: random forest hyper-parameters
 */
void validity_check(const RF_params rf_params) {
  ASSERT((rf_params.n_trees > 0), "Invalid n_trees %d", rf_params.n_trees);
  ASSERT((rf_params.max_samples > 0) && (rf_params.max_samples <= 1.0),
         "max_samples value %f outside permitted (0, 1] range",
         rf_params.max_samples);
  DecisionTree::validity_check(rf_params.tree_params);
}

/**
 * @brief Print all random forest hyper-parameters.
 * @param[in] rf_params: random forest hyper-parameters
 */
void print(const RF_params rf_params) {
  ML::PatternSetter _("%v");
  CUML_LOG_DEBUG("n_trees: %d", rf_params.n_trees);
  CUML_LOG_DEBUG("bootstrap: %d", rf_params.bootstrap);
  CUML_LOG_DEBUG("max_samples: %f", rf_params.max_samples);
  CUML_LOG_DEBUG("n_streams: %d", rf_params.n_streams);
  DecisionTree::print(rf_params.tree_params);
}

/**
 * @brief Set the trees pointer of RandomForestMetaData to nullptr.
 * @param[in, out] forest: CPU pointer to RandomForestMetaData.
 */
template <class T, class L>
void null_trees_ptr(RandomForestMetaData<T, L>*& forest) {
  forest->trees = nullptr;
}

/**
 * @brief Deletes RandomForestMetaData object
 * @param[in] forest: CPU pointer to RandomForestMetaData.
 */
template <class T, class L>
void delete_rf_metadata(RandomForestMetaData<T, L>* forest) {
  delete forest;
}

template <class T, class L>
std::string _get_rf_text(const RandomForestMetaData<T, L>* forest,
                         bool summary) {
  ML::PatternSetter _("%v");
  if (!forest || !forest->trees) {
    return "Empty forest";
  } else {
    std::ostringstream oss;
    oss << "Forest has " << forest->rf_params.n_trees << " trees, "
        << "max_depth " << forest->rf_params.tree_params.max_depth
        << ", and max_leaves " << forest->rf_params.tree_params.max_leaves
        << "\n";
    for (int i = 0; i < forest->rf_params.n_trees; i++) {
      oss << "Tree #" << i << "\n";
      if (summary) {
        oss << DecisionTree::get_tree_summary_text<T, L>(&(forest->trees[i]))
            << "\n";
      } else {
        oss << DecisionTree::get_tree_text<T, L>(&(forest->trees[i])) << "\n";
      }
    }
    return oss.str();
  }
}

template <class T, class L>
std::string _get_rf_json(const RandomForestMetaData<T, L>* forest) {
  if (!forest || !forest->trees) {
    return "[]";
  }
  std::ostringstream oss;
  oss << "[\n";
  for (int i = 0; i < forest->rf_params.n_trees; i++) {
    oss << DecisionTree::get_tree_json<T, L>(&(forest->trees[i]));
    if (i < forest->rf_params.n_trees - 1) {
      oss << ",\n";
    }
  }
  oss << "\n]";
  return oss.str();
}

/**
 * @brief Print summary for all trees in the random forest.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 * @param[in] forest: CPU pointer to RandomForestMetaData struct.
 */
template <class T, class L>
std::string get_rf_summary_text(const RandomForestMetaData<T, L>* forest) {
  return _get_rf_text(forest, true);
}

/**
 * @brief Print detailed view of all trees in the random forest.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 * @param[in] forest: CPU pointer to RandomForestMetaData struct.
 */
template <class T, class L>
std::string get_rf_detailed_text(const RandomForestMetaData<T, L>* forest) {
  return _get_rf_text(forest, false);
}

template <class T, class L>
std::string get_rf_json(const RandomForestMetaData<T, L>* forest) {
  return _get_rf_json(forest);
}

template <class T, class L>
void build_treelite_forest(ModelHandle* model_handle,
                           const RandomForestMetaData<T, L>* forest,
                           int num_features, int task_category) {
  auto parent_model = tl::Model::Create<T, T>();
  tl::ModelImpl<T, T>* model =
    dynamic_cast<tl::ModelImpl<T, T>*>(parent_model.get());
  ASSERT(model != nullptr, "Invalid downcast to tl::ModelImpl");

  unsigned int num_class;
  if (task_category > 2) {
    // Multi-class classification
    num_class = task_category;
    model->task_type = tl::TaskType::kMultiClfProbDistLeaf;
    std::strcpy(model->param.pred_transform, "max_index");
  } else {
    // Binary classification or regression
    num_class = 1;
    model->task_type = tl::TaskType::kBinaryClfRegr;
  }

  model->task_param = tl::TaskParameter{tl::TaskParameter::OutputType::kFloat,
                                        false, num_class, num_class};
  model->num_feature = num_features;
  model->average_tree_output = true;
  model->SetTreeLimit(forest->rf_params.n_trees);

  std::vector<Node_ID_info<T, L>> working_queue_1;
  std::vector<Node_ID_info<T, L>> working_queue_2;

#pragma omp parallel for private(working_queue_1, working_queue_2)
  for (int i = 0; i < forest->rf_params.n_trees; i++) {
    DecisionTree::TreeMetaDataNode<T, L>& rf_tree = forest->trees[i];

    if (rf_tree.sparsetree.size() != 0) {
      model->trees[i] = DecisionTree::build_treelite_tree<T, L>(
        rf_tree, num_class, working_queue_1, working_queue_2);
    }
  }

  *model_handle = static_cast<ModelHandle>(parent_model.release());
}

/**
 * @brief Compares the trees present in concatenated treelite forest with the trees
 *   of the forests present in the different workers. If there is a difference in the two
 *   then an error statement will be thrown.
 * @param[in] tree_from_concatenated_forest: Tree info from the concatenated forest.
 * @param[in] tree_from_individual_forest: Tree info from the forest present in each worker.
 */
template <class T, class L>
void compare_trees(tl::Tree<T, L>& tree_from_concatenated_forest,
                   tl::Tree<T, L>& tree_from_individual_forest) {
  ASSERT(tree_from_concatenated_forest.num_nodes ==
           tree_from_individual_forest.num_nodes,
         "Error! Mismatch the number of nodes present in a tree in the "
         "concatenated forest and"
         " the tree present in the individual forests");
  for (int each_node = 0; each_node < tree_from_concatenated_forest.num_nodes;
       each_node++) {
    ASSERT(tree_from_concatenated_forest.IsLeaf(each_node) ==
             tree_from_individual_forest.IsLeaf(each_node),
           "Error! mismatch in the position of a leaf between concatenated "
           "forest and the"
           " individual forests ");
    ASSERT(tree_from_concatenated_forest.LeafValue(each_node) ==
             tree_from_individual_forest.LeafValue(each_node),
           "Error! leaf value mismatch between concatenated forest and the"
           " individual forests ");
    ASSERT(tree_from_concatenated_forest.RightChild(each_node) ==
             tree_from_individual_forest.RightChild(each_node),
           "Error! mismatch in the position of the node between concatenated "
           "forest and the"
           " individual forests ");
    ASSERT(tree_from_concatenated_forest.LeftChild(each_node) ==
             tree_from_individual_forest.LeftChild(each_node),
           "Error! mismatch in the position of the node between concatenated "
           "forest and the"
           " individual forests ");
    ASSERT(
      tree_from_concatenated_forest.SplitIndex(each_node) ==
        tree_from_individual_forest.SplitIndex(each_node),
      "Error! split index value mismatch between concatenated forest and the"
      " individual forests ");
  }
}

/**
 * @brief Compares the concatenated treelite model with the information of the forest
 *   present in the different workers. If there is a difference in the two then an error
 *   statement will be thrown.
 * @param[in] concat_tree_handle: ModelHandle for the concatenated forest.
 * @param[in] treelite_handles: List containing ModelHandles for the forest present in
 *   each worker.
 */
void compare_concat_forest_to_subforests(
  ModelHandle concat_tree_handle, std::vector<ModelHandle> treelite_handles) {
  size_t concat_forest;
  size_t total_num_trees = 0;
  for (int forest_idx = 0; forest_idx < treelite_handles.size(); forest_idx++) {
    size_t num_trees_each_forest;
    TREELITE_CHECK(TreeliteQueryNumTree(treelite_handles[forest_idx],
                                        &num_trees_each_forest));
    total_num_trees = total_num_trees + num_trees_each_forest;
  }

  TREELITE_CHECK(TreeliteQueryNumTree(concat_tree_handle, &concat_forest));

  ASSERT(
    concat_forest == total_num_trees,
    "Error! the number of trees in the concatenated forest and the sum "
    "of the trees present in the forests present in each worker are not equal");

  int concat_mod_tree_num = 0;
  tl::Model& concat_model = *(tl::Model*)(concat_tree_handle);
  for (int forest_idx = 0; forest_idx < treelite_handles.size(); forest_idx++) {
    tl::Model& model = *(tl::Model*)(treelite_handles[forest_idx]);

    ASSERT(
      concat_model.GetThresholdType() == model.GetThresholdType(),
      "Error! Concatenated forest does not have the same threshold type as "
      "the individual forests");
    ASSERT(
      concat_model.GetLeafOutputType() == model.GetLeafOutputType(),
      "Error! Concatenated forest does not have the same leaf output type as "
      "the individual forests");
    ASSERT(
      concat_model.num_feature == model.num_feature,
      "Error! number of features mismatch between concatenated forest and the"
      " individual forests");
    ASSERT(concat_model.task_param.num_class == model.task_param.num_class,
           "Error! number of classes mismatch between concatenated forest "
           "and the individual forests ");
    ASSERT(concat_model.average_tree_output == model.average_tree_output,
           "Error! average_tree_output flag value mismatch between "
           "concatenated forest and the individual forests");

    model.Dispatch([&concat_mod_tree_num, &concat_model](auto& model_inner) {
      // model_inner is of the concrete type tl::ModelImpl<T, L>
      using model_type = std::remove_reference_t<decltype(model_inner)>;
      auto& concat_model_inner = dynamic_cast<model_type&>(concat_model);
      for (int indiv_trees = 0; indiv_trees < model_inner.trees.size();
           indiv_trees++) {
        compare_trees(
          concat_model_inner.trees[concat_mod_tree_num + indiv_trees],
          model_inner.trees[indiv_trees]);
      }
      concat_mod_tree_num = concat_mod_tree_num + model_inner.trees.size();
    });
  }
}

/**
 * @brief Concatenates the forest information present in different workers to
 *  create a single forest. This concatenated forest is stored in a new treelite model.
 *  The model created is owned by and must be freed by the user.
 * @param[in] concat_tree_handle: ModelHandle for the concatenated forest.
 * @param[in] treelite_handles: List containing ModelHandles for the forest present in
 *   each worker.
 */
ModelHandle concatenate_trees(std::vector<ModelHandle> treelite_handles) {
  tl::Model& first_model = *(tl::Model*)treelite_handles[0];
  tl::Model* concat_model =
    first_model.Dispatch([&treelite_handles](auto& first_model_inner) {
      // first_model_inner is of the concrete type tl::ModelImpl<T, L>
      using model_type = std::remove_reference_t<decltype(first_model_inner)>;
      auto* concat_model = dynamic_cast<model_type*>(
        tl::Model::Create(first_model_inner.GetThresholdType(),
                          first_model_inner.GetLeafOutputType())
          .release());
      for (int forest_idx = 0; forest_idx < treelite_handles.size();
           forest_idx++) {
        tl::Model& model = *(tl::Model*)treelite_handles[forest_idx];
        auto& model_inner = dynamic_cast<model_type&>(model);
        for (const auto& tree : model_inner.trees) {
          concat_model->trees.push_back(tree.Clone());
        }
      }
      concat_model->num_feature = first_model_inner.num_feature;
      concat_model->task_param = first_model_inner.task_param;
      concat_model->average_tree_output = first_model_inner.average_tree_output;
      concat_model->param = first_model_inner.param;
      return static_cast<tl::Model*>(concat_model);
    });
  return concat_model;
}

/**
 * @defgroup RandomForestClassificationFit Random Forest Classification - Fit function
 * @brief Build (i.e., fit, train) random forest classifier for input data.
 * @param[in] user_handle: raft::handle_t
 * @param[in,out] forest: CPU pointer to RandomForestMetaData object. User allocated.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format,
 *   excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (int only), with one label per
 *   training sample. Device pointer.
 *   Assumption: labels were preprocessed to map to ascending numbers from 0;
 *   needed for current gini impl. in decision tree
 * @param[in] n_unique_labels: #unique label values (known during preprocessing)
 * @param[in] rf_params: Random Forest training hyper parameter struct.
 * @param[in] verbosity: verbosity level for logging messages during execution
 * @{
 */
void fit(const raft::handle_t& user_handle, RandomForestClassifierF*& forest,
         float* input, int n_rows, int n_cols, int* labels, int n_unique_labels,
         RF_params rf_params, int verbosity) {
  ML::PUSH_RANGE("RF::fit @randomforest.cu");
  ML::Logger::get().setLevel(verbosity);
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<float, int>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfClassifier<float>> rf_classifier =
    std::make_shared<rfClassifier<float>>(rf_params);
  rf_classifier->fit(user_handle, input, n_rows, n_cols, labels,
                     n_unique_labels, forest);
  ML::POP_RANGE();
}

void fit(const raft::handle_t& user_handle, RandomForestClassifierD*& forest,
         double* input, int n_rows, int n_cols, int* labels,
         int n_unique_labels, RF_params rf_params, int verbosity) {
  ML::PUSH_RANGE("RF::fit @randomforest.cu");
  ML::Logger::get().setLevel(verbosity);
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<double, int>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfClassifier<double>> rf_classifier =
    std::make_shared<rfClassifier<double>>(rf_params);
  rf_classifier->fit(user_handle, input, n_rows, n_cols, labels,
                     n_unique_labels, forest);
  ML::POP_RANGE();
}
/** @} */

/**
 * @defgroup RandomForestClassificationPredict Random Forest Classification - Predict function
 * @brief Predict target feature for input data; n-ary classification for
     single feature supported.
 * @param[in] user_handle: raft::handle_t.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbosity: verbosity level for logging messages during execution
 * @{
 */
void predict(const raft::handle_t& user_handle,
             const RandomForestClassifierF* forest, const float* input,
             int n_rows, int n_cols, int* predictions, int verbosity) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<float>> rf_classifier =
    std::make_shared<rfClassifier<float>>(forest->rf_params);
  rf_classifier->predict(user_handle, input, n_rows, n_cols, predictions,
                         forest, verbosity);
}

void predict(const raft::handle_t& user_handle,
             const RandomForestClassifierD* forest, const double* input,
             int n_rows, int n_cols, int* predictions, int verbosity) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<double>> rf_classifier =
    std::make_shared<rfClassifier<double>>(forest->rf_params);
  rf_classifier->predict(user_handle, input, n_rows, n_cols, predictions,
                         forest, verbosity);
}
/** @} */

/**
 * @addtogroup RandomForestClassificationPredict
 * @brief Predict target feature for input data; n-ary classification for
     single feature supported.
 * @param[in] user_handle: raft::handle_t.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbosity: verbosity level for logging messages during execution
 * @{
 */
void predictGetAll(const raft::handle_t& user_handle,
                   const RandomForestClassifierF* forest, const float* input,
                   int n_rows, int n_cols, int* predictions, int verbosity) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<float>> rf_classifier =
    std::make_shared<rfClassifier<float>>(forest->rf_params);
  rf_classifier->predictGetAll(user_handle, input, n_rows, n_cols, predictions,
                               forest, verbosity);
}

void predictGetAll(const raft::handle_t& user_handle,
                   const RandomForestClassifierD* forest, const double* input,
                   int n_rows, int n_cols, int* predictions, int verbosity) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfClassifier<double>> rf_classifier =
    std::make_shared<rfClassifier<double>>(forest->rf_params);
  rf_classifier->predictGetAll(user_handle, input, n_rows, n_cols, predictions,
                               forest, verbosity);
}
/** @} */

/**
 * @defgroup RandomForestClassificationScore Random Forest Classification - Score function
 * @brief Compare predicted features validate against ref_labels.
 * @param[in] user_handle: raft::handle_t.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbosity: verbosity level for logging messages during execution
 * @return RF_metrics struct with classification score (i.e., accuracy)
 * @{
 */
RF_metrics score(const raft::handle_t& user_handle,
                 const RandomForestClassifierF* forest, const int* ref_labels,
                 int n_rows, const int* predictions, int verbosity) {
  RF_metrics classification_score = rfClassifier<float>::score(
    user_handle, ref_labels, n_rows, predictions, verbosity);
  return classification_score;
}

RF_metrics score(const raft::handle_t& user_handle,
                 const RandomForestClassifierD* forest, const int* ref_labels,
                 int n_rows, const int* predictions, int verbosity) {
  RF_metrics classification_score = rfClassifier<double>::score(
    user_handle, ref_labels, n_rows, predictions, verbosity);
  return classification_score;
}

RF_params set_rf_params(int max_depth, int max_leaves, float max_features,
                        int n_bins, int split_algo, int min_samples_leaf,
                        int min_samples_split, float min_impurity_decrease,
                        bool bootstrap_features, bool bootstrap, int n_trees,
                        float max_samples, uint64_t seed,
                        CRITERION split_criterion, bool quantile_per_tree,
                        int cfg_n_streams, bool use_experimental_backend,
                        int max_batch_size) {
  DecisionTree::DecisionTreeParams tree_params;
  DecisionTree::set_tree_params(
    tree_params, max_depth, max_leaves, max_features, n_bins, split_algo,
    min_samples_leaf, min_samples_split, min_impurity_decrease,
    bootstrap_features, split_criterion, quantile_per_tree,
    use_experimental_backend, max_batch_size);
  RF_params rf_params;
  rf_params.n_trees = n_trees;
  rf_params.bootstrap = bootstrap;
  rf_params.max_samples = max_samples;
  rf_params.seed = seed;
  rf_params.n_streams = min(cfg_n_streams, omp_get_max_threads());
  if (n_trees < rf_params.n_streams) rf_params.n_streams = n_trees;
  rf_params.tree_params = tree_params;
  return rf_params;
}

/** @} */

/**
 * @defgroup RandomForestRegressorFit Random Forest Regression - Fit function
 * @brief Build (i.e., fit, train) random forest regressor for input data.
 * @param[in] user_handle: raft::handle_t
 * @param[in,out] forest: CPU pointer to RandomForestMetaData object. User allocated.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format,
 *   excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (float or double), with one label per
 *   training sample. Device pointer.
 * @param[in] rf_params: Random Forest training hyper parameter struct.
 * @param[in] verbosity: verbosity level for logging messages during execution
 * @{
 */
void fit(const raft::handle_t& user_handle, RandomForestRegressorF*& forest,
         float* input, int n_rows, int n_cols, float* labels,
         RF_params rf_params, int verbosity) {
  ML::PUSH_RANGE("RF::fit @randomforest.cu");
  ML::Logger::get().setLevel(verbosity);
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<float, float>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfRegressor<float>> rf_regressor =
    std::make_shared<rfRegressor<float>>(rf_params);
  rf_regressor->fit(user_handle, input, n_rows, n_cols, labels, forest);
  ML::POP_RANGE();
}

void fit(const raft::handle_t& user_handle, RandomForestRegressorD*& forest,
         double* input, int n_rows, int n_cols, double* labels,
         RF_params rf_params, int verbosity) {
  ML::PUSH_RANGE("RF::fit @randomforest.cu");
  ML::Logger::get().setLevel(verbosity);
  ASSERT(!forest->trees, "Cannot fit an existing forest.");
  forest->trees =
    new DecisionTree::TreeMetaDataNode<double, double>[rf_params.n_trees];
  forest->rf_params = rf_params;

  std::shared_ptr<rfRegressor<double>> rf_regressor =
    std::make_shared<rfRegressor<double>>(rf_params);
  rf_regressor->fit(user_handle, input, n_rows, n_cols, labels, forest);
  ML::POP_RANGE();
}
/** @} */

/**
 * @defgroup RandomForestRegressorPredict Random Forest Regression - Predict function
 * @brief Predict target feature for input data; regression for single feature supported.
 * @param[in] user_handle: raft::handle_t.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbosity: verbosity level for logging messages during execution
 * @{
 */
void predict(const raft::handle_t& user_handle,
             const RandomForestRegressorF* forest, const float* input,
             int n_rows, int n_cols, float* predictions, int verbosity) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfRegressor<float>> rf_regressor =
    std::make_shared<rfRegressor<float>>(forest->rf_params);
  rf_regressor->predict(user_handle, input, n_rows, n_cols, predictions, forest,
                        verbosity);
}

void predict(const raft::handle_t& user_handle,
             const RandomForestRegressorD* forest, const double* input,
             int n_rows, int n_cols, double* predictions, int verbosity) {
  ASSERT(forest->trees, "Cannot predict! No trees in the forest.");
  std::shared_ptr<rfRegressor<double>> rf_regressor =
    std::make_shared<rfRegressor<double>>(forest->rf_params);
  rf_regressor->predict(user_handle, input, n_rows, n_cols, predictions, forest,
                        verbosity);
}
/** @} */

/**
 * @defgroup RandomForestRegressorScore Random Forest Regression - Score function
 * @brief Predict target feature for input data and validate against ref_labels.
 * @param[in] user_handle: raft::handle_t.
 * @param[in] forest: CPU pointer to RandomForestMetaData object.
 *   The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbosity: verbosity level for logging messages during execution
 * @return RF_metrics struct with regression score (i.e., mean absolute error,
 *   mean squared error, median absolute error)
 * @{
 */
RF_metrics score(const raft::handle_t& user_handle,
                 const RandomForestRegressorF* forest, const float* ref_labels,
                 int n_rows, const float* predictions, int verbosity) {
  RF_metrics regression_score = rfRegressor<float>::score(
    user_handle, ref_labels, n_rows, predictions, verbosity);

  return regression_score;
}

RF_metrics score(const raft::handle_t& user_handle,
                 const RandomForestRegressorD* forest, const double* ref_labels,
                 int n_rows, const double* predictions, int verbosity) {
  RF_metrics regression_score = rfRegressor<double>::score(
    user_handle, ref_labels, n_rows, predictions, verbosity);
  return regression_score;
}
/** @} */

// Functions' specializations
template std::string get_rf_summary_text<float, int>(
  const RandomForestClassifierF* forest);
template std::string get_rf_summary_text<double, int>(
  const RandomForestClassifierD* forest);
template std::string get_rf_summary_text<float, float>(
  const RandomForestRegressorF* forest);
template std::string get_rf_summary_text<double, double>(
  const RandomForestRegressorD* forest);

template std::string get_rf_detailed_text<float, int>(
  const RandomForestClassifierF* forest);
template std::string get_rf_detailed_text<double, int>(
  const RandomForestClassifierD* forest);
template std::string get_rf_detailed_text<float, float>(
  const RandomForestRegressorF* forest);
template std::string get_rf_detailed_text<double, double>(
  const RandomForestRegressorD* forest);

template std::string get_rf_json<float, int>(
  const RandomForestClassifierF* forest);
template std::string get_rf_json<double, int>(
  const RandomForestClassifierD* forest);
template std::string get_rf_json<float, float>(
  const RandomForestRegressorF* forest);
template std::string get_rf_json<double, double>(
  const RandomForestRegressorD* forest);

template void null_trees_ptr<float, int>(RandomForestClassifierF*& forest);
template void null_trees_ptr<double, int>(RandomForestClassifierD*& forest);
template void null_trees_ptr<float, float>(RandomForestRegressorF*& forest);
template void null_trees_ptr<double, double>(RandomForestRegressorD*& forest);

template void delete_rf_metadata<float, int>(RandomForestClassifierF* forest);
template void delete_rf_metadata<double, int>(RandomForestClassifierD* forest);
template void delete_rf_metadata<float, float>(RandomForestRegressorF* forest);
template void delete_rf_metadata<double, double>(
  RandomForestRegressorD* forest);

template void build_treelite_forest<float, int>(
  ModelHandle* model, const RandomForestMetaData<float, int>* forest,
  int num_features, int task_category);
template void build_treelite_forest<double, int>(
  ModelHandle* model, const RandomForestMetaData<double, int>* forest,
  int num_features, int task_category);
template void build_treelite_forest<float, float>(
  ModelHandle* model, const RandomForestMetaData<float, float>* forest,
  int num_features, int task_category);
template void build_treelite_forest<double, double>(
  ModelHandle* model, const RandomForestMetaData<double, double>* forest,
  int num_features, int task_category);
}  // End namespace ML
