#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <GPUTreeShap/gpu_treeshap.h>
#include <algorithm>
#include <cstddef>
#include <cstdint>
#include <cuml/explainer/tree_shap.hpp>
#include <iostream>
#include <limits>
#include <memory>
#include <raft/error.hpp>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <treelite/tree.h>
#include <type_traits>
#include <vector>

namespace tl = treelite;

/* All functions and classes defined in this anonymous namespace are strictly
 * for internal use by GPUTreeSHAP. */
namespace {

// A poor man's Span class.
// TODO(hcho3): Remove this class once RAFT implements a span abstraction.
template <typename T>
class Span {
 private:
  T* ptr_{nullptr};
  std::size_t size_{0};

 public:
  Span() = default;
  __host__ __device__ Span(T* ptr, std::size_t size) : ptr_(ptr), size_(size) {}
  __host__ explicit Span(std::vector<T>& vec) : ptr_(vec.data()), size_(vec.size()) {}
  __host__ explicit Span(thrust::device_vector<T>& vec)
    : ptr_(thrust::raw_pointer_cast(vec.data())), size_(vec.size())
  {
  }
  __host__ __device__ Span(const Span& other) : ptr_(other.ptr_), size_(other.size_) {}
  __host__ __device__ Span(Span&& other) : ptr_(other.ptr_), size_(other.size_)
  {
    other.ptr_  = nullptr;
    other.size_ = 0;
  }
  __host__ __device__ ~Span() {}
  __host__ __device__ Span& operator=(const Span& other)
  {
    ptr_  = other.ptr_;
    size_ = other.size_;
    return *this;
  }
  __host__ __device__ Span& operator=(Span&& other)
  {
    ptr_        = other.ptr_;
    size_       = other.size_;
    other.ptr_  = nullptr;
    other.size_ = 0;
    return *this;
  }
  __host__ __device__ std::size_t Size() const { return size_; }
  __host__ __device__ T* Data() const { return ptr_; }
  __host__ __device__ T& operator[](std::size_t offset) const { return *(ptr_ + offset); }
  __host__ __device__ Span<T> Subspan(std::size_t offset, std::size_t count)
  {
    return Span{ptr_ + offset, count};
  }
};

// A poor man's bit field, to be used to account for categorical splits in SHAP computation
// Inspired by xgboost::BitFieldContainer
template <typename T>
class BitField {
 private:
  static std::size_t constexpr kValueSize = sizeof(T) * 8;
  static std::size_t constexpr kOne       = 1;  // force correct data type

  Span<T> bits_;

 public:
  BitField() = default;
  __host__ __device__ explicit BitField(Span<T> bits) : bits_(bits) {}
  __host__ __device__ BitField(const BitField& other) : bits_(other.bits_) {}
  BitField& operator=(const BitField& other) = default;
  BitField& operator=(BitField&& other) = default;
  __host__ __device__ bool Check(std::size_t pos) const
  {
    T bitmask = kOne << (pos % kValueSize);
    return static_cast<bool>(bits_[pos / kValueSize] & bitmask);
  }
  __host__ __device__ void Set(std::size_t pos)
  {
    T bitmask = kOne << (pos % kValueSize);
    bits_[pos / kValueSize] |= bitmask;
  }
  __host__ __device__ void Intersect(const BitField other)
  {
    if (bits_.Data() == other.bits_.Data()) { return; }
    std::size_t size = min(bits_.Size(), other.bits_.Size());
    for (std::size_t i = 0; i < size; ++i) {
      bits_[i] &= other.bits_[i];
    }
    if (bits_.Size() > size) {
      for (std::size_t i = size; i < bits_.Size(); ++i) {
        bits_[i] = 0;
      }
    }
  }
  __host__ __device__ std::size_t Size() const { return kValueSize * bits_.Size(); }
  __host__ static std::size_t ComputeStorageSize(std::size_t n_cat)
  {
    return n_cat / kValueSize + (n_cat % kValueSize != 0);
  }

  static_assert(!std::is_signed<T>::value, "Must use unsiged type as underlying storage.");
};

template <typename ThresholdType>
struct SplitCondition {
  SplitCondition() = default;
  SplitCondition(ThresholdType feature_lower_bound,
                 ThresholdType feature_upper_bound,
                 tl::Operator comparison_op)
    : feature_lower_bound(feature_lower_bound),
      feature_upper_bound(feature_upper_bound),
      comparison_op(comparison_op)
  {
    if (feature_lower_bound > feature_upper_bound) {
      RAFT_FAIL("Lower bound cannot exceed upper bound");
    }
    if (comparison_op != tl::Operator::kLT && comparison_op != tl::Operator::kLE &&
        comparison_op != tl::Operator::kNone) {
      RAFT_FAIL("Unsupported comparison operator");
    }
  }

  // Lower and upper bounds on feature values flowing down this path
  ThresholdType feature_lower_bound;
  ThresholdType feature_upper_bound;
  // Comparison operator used in the test. For now only < (kLT) and <= (kLE)
  // are supported.
  tl::Operator comparison_op;

  // Does this instance flow down this path?
  __host__ __device__ bool EvaluateSplit(ThresholdType x) const
  {
    if (comparison_op == tl::Operator::kLE) {
      return x > feature_lower_bound && x <= feature_upper_bound;
    }
    return x >= feature_lower_bound && x < feature_upper_bound;
  }

  // Combine two split conditions on the same feature
  __host__ __device__ void Merge(const SplitCondition& other)
  {  // Combine duplicate features
    feature_lower_bound = max(feature_lower_bound, other.feature_lower_bound);
    feature_upper_bound = min(feature_upper_bound, other.feature_upper_bound);
  }
  static_assert(std::is_same<ThresholdType, float>::value ||
                  std::is_same<ThresholdType, double>::value,
                "ThresholdType must be a float or double");
};

template <typename ThresholdType, typename LeafType>
struct PathSegmentExtractor {
  using PathElementT = gpu_treeshap::PathElement<SplitCondition<ThresholdType>>;
  std::vector<PathElementT>& path_segments;
  std::size_t& path_idx;

  static constexpr ThresholdType inf{std::numeric_limits<ThresholdType>::infinity()};

  PathSegmentExtractor(std::vector<PathElementT>& path_segments, std::size_t& path_idx)
    : path_segments(path_segments), path_idx(path_idx)
  {
  }

  void node_handler(const tl::Tree<ThresholdType, LeafType>& tree,
                    int child_idx,
                    int parent_idx,
                    int group_id,
                    float v)
  {
    constexpr auto inf   = std::numeric_limits<ThresholdType>::infinity();
    double zero_fraction = 1.0;
    bool has_count_info  = false;
    if (tree.HasSumHess(parent_idx) && tree.HasSumHess(child_idx)) {
      zero_fraction  = static_cast<double>(tree.SumHess(child_idx) / tree.SumHess(parent_idx));
      has_count_info = true;
    }
    if (!has_count_info && tree.HasDataCount(parent_idx) && tree.HasDataCount(child_idx)) {
      zero_fraction  = static_cast<double>(tree.DataCount(child_idx)) / tree.DataCount(parent_idx);
      has_count_info = true;
    }
    if (!has_count_info) { RAFT_FAIL("Tree model doesn't have data count information"); }
    // Encode the range of feature values that flow down this path
    bool is_left_path = tree.LeftChild(parent_idx) == child_idx;
    if (tree.SplitType(parent_idx) == tl::SplitFeatureType::kCategorical) {
      RAFT_FAIL(
        "Only trees with numerical splits are supported. "
        "Trees with categorical splits are not supported yet.");
    }
    ThresholdType lower_bound = is_left_path ? -inf : tree.Threshold(parent_idx);
    ThresholdType upper_bound = is_left_path ? tree.Threshold(parent_idx) : inf;
    auto comparison_op        = tree.ComparisonOp(parent_idx);
    path_segments.push_back(gpu_treeshap::PathElement<SplitCondition<ThresholdType>>{
      path_idx,
      tree.SplitIndex(parent_idx),
      group_id,
      SplitCondition{lower_bound, upper_bound, comparison_op},
      zero_fraction,
      v});
  }

  void root_handler(const tl::Tree<ThresholdType, LeafType>& tree,
                    int child_idx,
                    int group_id,
                    float v)
  {
    // Root node has feature -1
    auto comparison_op = tree.ComparisonOp(child_idx);
    path_segments.push_back(gpu_treeshap::PathElement<SplitCondition<ThresholdType>>{
      path_idx, -1, group_id, SplitCondition{-inf, inf, comparison_op}, 1.0, v});
  }

  void new_path_handler() { ++path_idx; }
};

template <typename ThresholdType>
class TreePathInfoImpl : public ML::Explainer::TreePathInfo {
 public:
  ThresholdTypeEnum threshold_type;
  int num_tree;
  float global_bias;
  tl::TaskType task_type;
  tl::TaskParam task_param;
  bool average_tree_output;
  std::vector<gpu_treeshap::PathElement<SplitCondition<ThresholdType>>> paths;

  static_assert(std::is_same<ThresholdType, float>::value ||
                  std::is_same<ThresholdType, double>::value,
                "ThresholdType must be a float or double");

  TreePathInfoImpl()
  {
    if constexpr (std::is_same<ThresholdType, double>::value) {
      threshold_type = ThresholdTypeEnum::kDouble;
    } else {
      threshold_type = ThresholdTypeEnum::kFloat;
    }
  }
  virtual ~TreePathInfoImpl() = default;

  ThresholdTypeEnum GetThresholdType() const override { return threshold_type; }
};

class DenseDatasetWrapper {
  const float* data;
  std::size_t num_rows;
  std::size_t num_cols;

 public:
  DenseDatasetWrapper() = default;
  DenseDatasetWrapper(const float* data, int num_rows, int num_cols)
    : data(data), num_rows(num_rows), num_cols(num_cols)
  {
  }
  __device__ float GetElement(std::size_t row_idx, std::size_t col_idx) const
  {
    return data[row_idx * num_cols + col_idx];
  }
  __host__ __device__ std::size_t NumRows() const { return num_rows; }
  __host__ __device__ std::size_t NumCols() const { return num_cols; }
};

template <typename ThresholdType>
void gpu_treeshap_impl(const TreePathInfoImpl<ThresholdType>* path_info,
                       const float* data,
                       std::size_t n_rows,
                       std::size_t n_cols,
                       float* out_preds)
{
  DenseDatasetWrapper X(data, n_rows, n_cols);

  std::size_t num_groups = 1;
  if (path_info->task_param.num_class > 1) {
    num_groups = static_cast<std::size_t>(path_info->task_param.num_class);
  }
  std::size_t pred_size = n_rows * num_groups * (n_cols + 1);

  thrust::device_ptr<float> out_preds_ptr = thrust::device_pointer_cast(out_preds);
  gpu_treeshap::GPUTreeShap(X,
                            path_info->paths.begin(),
                            path_info->paths.end(),
                            num_groups,
                            out_preds_ptr,
                            out_preds_ptr + pred_size);

  // Post-processing
  auto count_iter  = thrust::make_counting_iterator(0);
  auto num_tree    = path_info->num_tree;
  auto global_bias = path_info->global_bias;
  if (path_info->average_tree_output) {
    thrust::for_each(
      thrust::device, count_iter, count_iter + pred_size, [=] __device__(std::size_t idx) {
        out_preds[idx] /= num_tree;
      });
  }
  thrust::for_each(
    thrust::device,
    count_iter,
    count_iter + (n_rows * num_groups),
    [=] __device__(std::size_t idx) { out_preds[(idx + 1) * (n_cols + 1) - 1] += global_bias; });
}

}  // anonymous namespace

namespace ML {
namespace Explainer {
// Traverse a path from the root node to a leaf node and call the handler functions for each node.
// The fields group_id and v (leaf value) will be passed to the handler.
template <typename ThresholdType, typename LeafType, typename PathHandler>
void traverse_towards_leaf_node(const tl::Tree<ThresholdType, LeafType>& tree,
                                int leaf_node_id,
                                int group_id,
                                float v,
                                const std::vector<int>& parent_id,
                                PathHandler& path_handler)
{
  int child_idx  = leaf_node_id;
  int parent_idx = parent_id[child_idx];
  while (parent_idx != -1) {
    path_handler.node_handler(tree, child_idx, parent_idx, group_id, v);
    child_idx  = parent_idx;
    parent_idx = parent_id[child_idx];
  }
  path_handler.root_handler(tree, child_idx, group_id, v);
}

// Visit every path segments in a single tree and call handler functions for each segment.
template <typename ThresholdType, typename LeafType, typename PathHandler>
void visit_path_segments_in_tree(const std::vector<tl::Tree<ThresholdType, LeafType>>& tree_list,
                                 std::size_t tree_idx,
                                 bool use_vector_leaf,
                                 int num_groups,
                                 PathHandler& path_handler)
{
  if (num_groups < 1) { RAFT_FAIL("num_groups must be at least 1"); }

  const tl::Tree<ThresholdType, LeafType>& tree = tree_list[tree_idx];

  // Compute parent ID of each node
  std::vector<int> parent_id(tree.num_nodes, -1);
  for (int i = 0; i < tree.num_nodes; i++) {
    if (!tree.IsLeaf(i)) {
      parent_id[tree.LeftChild(i)]  = i;
      parent_id[tree.RightChild(i)] = i;
    }
  }

  for (int nid = 0; nid < tree.num_nodes; nid++) {
    if (tree.IsLeaf(nid)) {  // For each leaf node...
      // Extract path segments by traversing the path from the leaf node to the root node
      // If use_vector_leaf=True, repeat the path segments N times, where N = num_groups
      if (use_vector_leaf) {
        auto leaf_vector = tree.LeafVector(nid);
        if (leaf_vector.size() != static_cast<std::size_t>(num_groups)) {
          RAFT_FAIL("Expected leaf vector of length %d but got %d instead",
                    num_groups,
                    static_cast<int>(leaf_vector.size()));
        }
        for (int group_id = 0; group_id < num_groups; ++group_id) {
          traverse_towards_leaf_node(
            tree, nid, group_id, leaf_vector[group_id], parent_id, path_handler);
          path_handler.new_path_handler();
        }
      } else {
        int group_id    = static_cast<int>(tree_idx) % num_groups;
        auto leaf_value = tree.LeafValue(nid);
        traverse_towards_leaf_node(tree, nid, group_id, leaf_value, parent_id, path_handler);
        path_handler.new_path_handler();
      }
    }
  }
}

// Visit every path segments in the whole tree ensemble model
template <typename ThresholdType, typename LeafType, typename PathHandler>
void visit_path_segments_in_model(const tl::ModelImpl<ThresholdType, LeafType>& model,
                                  PathHandler& path_handler)
{
  int num_groups = 1;
  bool use_vector_leaf;
  if (model.task_param.num_class > 1) { num_groups = model.task_param.num_class; }
  if (model.task_type == tl::TaskType::kBinaryClfRegr ||
      model.task_type == tl::TaskType::kMultiClfGrovePerClass) {
    use_vector_leaf = false;
  } else if (model.task_type == tl::TaskType::kMultiClfProbDistLeaf) {
    use_vector_leaf = true;
  } else {
    RAFT_FAIL("Unsupported task_type: %d", static_cast<int>(model.task_type));
  }

  for (std::size_t tree_idx = 0; tree_idx < model.trees.size(); ++tree_idx) {
    visit_path_segments_in_tree(model.trees, tree_idx, use_vector_leaf, num_groups, path_handler);
  }
}

template <typename ThresholdType, typename LeafType>
std::unique_ptr<TreePathInfo> extract_path_info_impl(
  const tl::ModelImpl<ThresholdType, LeafType>& model)
{
  if (!std::is_same<ThresholdType, LeafType>::value) {
    RAFT_FAIL("ThresholdType and LeafType must be identical");
  }
  if (!std::is_same<ThresholdType, float>::value && !std::is_same<ThresholdType, double>::value) {
    RAFT_FAIL("ThresholdType must be either float32 or float64");
  }

  std::unique_ptr<TreePathInfo> path_info_ptr = std::make_unique<TreePathInfoImpl<ThresholdType>>();
  auto* path_info = dynamic_cast<TreePathInfoImpl<ThresholdType>*>(path_info_ptr.get());

  // Each path segment will have path_idx field, which uniquely identifies the path to which the
  // segment belongs.
  std::size_t path_idx = 0;
  PathSegmentExtractor<ThresholdType, LeafType> path_extractor{path_info->paths, path_idx};
  visit_path_segments_in_model(model, path_extractor);

  path_info->global_bias         = model.param.global_bias;
  path_info->task_type           = model.task_type;
  path_info->task_param          = model.task_param;
  path_info->average_tree_output = model.average_tree_output;
  path_info->num_tree            = static_cast<int>(model.trees.size());

  return path_info_ptr;
}

std::unique_ptr<TreePathInfo> extract_path_info(ModelHandle model)
{
  const tl::Model& model_ref = *static_cast<tl::Model*>(model);

  return model_ref.Dispatch([&](const auto& model_inner) {
    // model_inner is of the concrete type tl::ModelImpl<threshold_t, leaf_t>
    return extract_path_info_impl(model_inner);
  });
}

void gpu_treeshap(const TreePathInfo* path_info,
                  const float* data,
                  std::size_t n_rows,
                  std::size_t n_cols,
                  float* out_preds)
{
  switch (path_info->GetThresholdType()) {
    case TreePathInfo::ThresholdTypeEnum::kDouble: {
      const auto* path_info_casted = dynamic_cast<const TreePathInfoImpl<double>*>(path_info);
      gpu_treeshap_impl(path_info_casted, data, n_rows, n_cols, out_preds);
    } break;
    case TreePathInfo::ThresholdTypeEnum::kFloat:
    default: {
      const auto* path_info_casted = dynamic_cast<const TreePathInfoImpl<float>*>(path_info);
      gpu_treeshap_impl(path_info_casted, data, n_rows, n_cols, out_preds);
    } break;
  }
}

}  // namespace Explainer
}  // namespace ML
