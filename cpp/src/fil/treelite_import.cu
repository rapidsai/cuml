#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file treelite_import.cu converts from treelite format to a FIL-centric CPU-RAM format, so that
 * fil.cu can make a `forest` object out of it. */

#include "common.cuh"    // for node_traits, num_trees, tree_num_nodes
#include "internal.cuh"  // for MAX_FIL_INT_FLOAT, BITS_PER_BYTE, cat_feature_counters, cat_sets, cat_sets_owner, categorical_sets, leaf_algo_t

#include <cuml/common/logger.hpp>  // for CUML_LOG_WARN
#include <cuml/fil/fil.h>  // for algo_t, from_treelite, storage_type_repr, storage_type_t, treelite_params_t
#include <cuml/fil/fnv_hash.h>  // for fowler_noll_vo_fingerprint64_32

#include <raft/cudart_utils.h>  // for RAFT_CUDA_TRY
#include <raft/error.hpp>       // for ASSERT
#include <raft/handle.hpp>      // for handle_t

#include <treelite/base.h>   // for Operator, SplitFeatureType, kGE, kGT, kLE, kLT, kNumerical
#include <treelite/c_api.h>  // for ModelHandle
#include <treelite/tree.h>   // for Tree, Model, ModelImpl, ModelParam

#include <omp.h>  // for omp

#include <algorithm>    // for std::max
#include <bitset>       // for std::bitset
#include <cmath>        // for NAN
#include <cstddef>      // for std::size_t
#include <cstdint>      // for uint8_t
#include <iosfwd>       // for ios, stringstream
#include <stack>        // for std::stack
#include <string>       // for std::string
#include <type_traits>  // for std::is_same

namespace ML {
namespace fil {

namespace tl = treelite;

std::ostream& operator<<(std::ostream& os, const cat_sets_owner& cso)
{
  os << "\nbits { ";
  for (uint8_t b : cso.bits) {
    os << std::bitset<BITS_PER_BYTE>(b) << " ";
  }
  os << " }\nmax_matching {";
  for (float fid_num_cats : cso.fid_num_cats) {
    os << static_cast<int>(fid_num_cats) - 1 << " ";
  }
  os << " }";
  return os;
}

template <typename T, typename L>
int tree_root(const tl::Tree<T, L>& tree)
{
  return 0;  // Treelite format assumes that the root is 0
}

// a no-op placeholder for values and callables alike
struct empty {
  template <typename... Args>
  void operator()(Args...)
  {
  }
};

/** walk a Treelite tree, visiting each inner node with visit_inner and each leaf node with
  visit_leaf. See walk_tree::element::state documentation for how TraversalState is retained
during traversal. Any per-tree state during traversal should be captured by the lambdas themselves.
  visit_inner(int node_id, TraversalState state) should return a pair of new states, one for
each child node. visit_leaf(int, TraversalState) returns nothing.
**/
template <typename T, typename L, typename InnerFunc, typename LeafFunc = empty>
inline void walk_tree(const tl::Tree<T, L>& tree,
                      InnerFunc visit_inner,
                      LeafFunc visit_leaf = empty())
{
  if constexpr (std::is_invocable<InnerFunc, int>()) {
    /// wrapper for empty path state
    walk_tree(
      tree,
      [&](int nid, empty val) {
        visit_inner(nid);
        return std::pair<empty, empty>();
      },
      [&](int nid, empty val) { visit_leaf(nid); });
  } else {
    using TraversalState = decltype(visit_inner(int(), {}).first);
    /// needed to visit a node
    struct element {
      int tl_node_id;
      /// Retained while visiting nodes on a single path from root to leaf.
      /// This generalizes the node index that's carried over during inference tree traversal.
      TraversalState state;
    };
    std::stack<element> stack;
    stack.push(element{tree_root(tree), TraversalState()});
    while (!stack.empty()) {
      element i = stack.top();
      stack.pop();
      while (!tree.IsLeaf(i.tl_node_id)) {
        auto [left_state, right_state] = visit_inner(i.tl_node_id, i.state);
        stack.push(element{tree.LeftChild(i.tl_node_id), left_state});
        i = element{tree.RightChild(i.tl_node_id), right_state};
      }
      visit_leaf(i.tl_node_id, i.state);
    }
  }
}

template <typename T, typename L>
inline int max_depth(const tl::Tree<T, L>& tree)
{
  int tree_depth = 0;
  walk_tree(
    tree,
    [](int node_id, int node_depth) {
      // trees of this depth aren't used, so it most likely means bad input data,
      // e.g. cycles in the forest
      constexpr int DEPTH_LIMIT = 500;
      ASSERT(node_depth < DEPTH_LIMIT, "node_depth limit reached, might be a cycle in the tree");
      return std::pair(node_depth + 1, node_depth + 1);
    },
    [&](int node_id, int node_depth) { tree_depth = std::max(node_depth, tree_depth); });
  return tree_depth;
}

template <typename T, typename L>
int max_depth(const tl::ModelImpl<T, L>& model)
{
  int depth         = 0;
  const auto& trees = model.trees;
#pragma omp parallel for reduction(max : depth)
  for (size_t i = 0; i < trees.size(); ++i) {
    const auto& tree = trees[i];
    depth            = std::max(depth, max_depth(tree));
  }
  return depth;
}

void elementwise_combine(std::vector<cat_feature_counters>& dst,
                         const std::vector<cat_feature_counters>& extra)
{
  std::transform(dst.begin(), dst.end(), extra.begin(), dst.begin(), cat_feature_counters::combine);
}

// constructs a vector of size n_cols (number of features, or columns) from a Treelite tree,
// where each feature has a maximum matching category and node count (from this tree alone).
template <typename T, typename L>
inline std::vector<cat_feature_counters> cat_counter_vec(const tl::Tree<T, L>& tree, int n_cols)
{
  std::vector<cat_feature_counters> res(n_cols);
  walk_tree(tree, [&](int node_id) {
    if (tree.SplitType(node_id) == tl::SplitFeatureType::kCategorical) {
      std::vector<std::uint32_t> mmv = tree.MatchingCategories(node_id);
      int max_matching_cat;
      if (mmv.size() > 0) {
        // in `struct cat_feature_counters` and GPU structures, int(max_matching_cat) is safe
        // because all precise int floats fit into ints, which are asserted to be 32 bits
        max_matching_cat = mmv.back();
        ASSERT(max_matching_cat <= MAX_FIL_INT_FLOAT,
               "FIL cannot infer on "
               "more than %d matching categories",
               MAX_FIL_INT_FLOAT);
      } else {
        max_matching_cat = -1;
      }
      cat_feature_counters& counters = res[tree.SplitIndex(node_id)];
      counters = cat_feature_counters::combine(counters, cat_feature_counters{max_matching_cat, 1});
    }
  });

  return res;
}

// computes overall categorical bit pool size for a tree imported from the Treelite tree
template <typename T, typename L>
inline std::size_t bit_pool_size(const tl::Tree<T, L>& tree, const categorical_sets& cat_sets)
{
  std::size_t size = 0;
  walk_tree(tree, [&](int node_id) {
    if (tree.SplitType(node_id) == tl::SplitFeatureType::kCategorical &&
        tree.MatchingCategories(node_id).size() > 0) {
      size += cat_sets.sizeof_mask(tree.SplitIndex(node_id));
    }
  });
  return size;
}

template <typename T, typename L>
cat_sets_owner allocate_cat_sets_owner(const tl::ModelImpl<T, L>& model)
{
#pragma omp declare reduction(cat_counter_vec_red : std::vector<cat_feature_counters> \
      : elementwise_combine(omp_out, omp_in))                 \
    initializer(omp_priv = omp_orig)
  const auto& trees = model.trees;
  cat_sets_owner cat_sets;
  std::vector<cat_feature_counters> counters(model.num_feature);
#pragma omp parallel for reduction(cat_counter_vec_red : counters)
  for (std::size_t i = 0; i < trees.size(); ++i) {
    elementwise_combine(counters, cat_counter_vec(trees[i], model.num_feature));
  }
  cat_sets.consume_counters(counters);
  std::vector<std::size_t> bit_pool_sizes(trees.size());
#pragma omp parallel for
  for (std::size_t i = 0; i < trees.size(); ++i) {
    bit_pool_sizes[i] = bit_pool_size(trees[i], cat_sets.accessor());
  }
  cat_sets.consume_bit_pool_sizes(bit_pool_sizes);
  return cat_sets;
}

void adjust_threshold(float* pthreshold, bool* swap_child_nodes, tl::Operator comparison_op)
{
  // in treelite (take left node if val [op] threshold),
  // the meaning of the condition is reversed compared to FIL;
  // thus, "<" in treelite corresonds to comparison ">=" used by FIL
  // https://github.com/dmlc/treelite/blob/master/include/treelite/tree.h#L243
  if (isnan(*pthreshold)) {
    *swap_child_nodes = !*swap_child_nodes;
    return;
  }
  switch (comparison_op) {
    case tl::Operator::kLT: break;
    case tl::Operator::kLE:
      // x <= y is equivalent to x < y', where y' is the next representable float
      *pthreshold = std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
      break;
    case tl::Operator::kGT:
      // x > y is equivalent to x >= y', where y' is the next representable float
      // left and right still need to be swapped
      *pthreshold = std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
    case tl::Operator::kGE:
      // swap left and right
      *swap_child_nodes = !*swap_child_nodes;
      break;
    default: ASSERT(false, "only <, >, <= and >= comparisons are supported");
  }
}

/** if the vector consists of zeros and a single one, return the position
for the one (assumed class label). Else, asserts false.
If the vector contains a NAN, asserts false */
template <typename L>
int find_class_label_from_one_hot(L* vector, int len)
{
  bool found_label = false;
  int out;
  for (int i = 0; i < len; ++i) {
    if (vector[i] == static_cast<L>(1.0)) {
      ASSERT(!found_label, "label vector contains multiple 1.0f");
      out         = i;
      found_label = true;
    } else {
      ASSERT(vector[i] == static_cast<L>(0.0),
             "label vector contains values other than 0.0 and 1.0");
    }
  }
  ASSERT(found_label, "did not find 1.0f in vector");
  return out;
}

template <typename fil_node_t, typename T, typename L>
void tl2fil_leaf_payload(fil_node_t* fil_node,
                         int fil_node_id,
                         const tl::Tree<T, L>& tl_tree,
                         int tl_node_id,
                         const forest_params_t& forest_params,
                         std::vector<float>* vector_leaf,
                         size_t* leaf_counter)
{
  auto vec = tl_tree.LeafVector(tl_node_id);
  switch (forest_params.leaf_algo) {
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(vec.size() == static_cast<std::size_t>(forest_params.num_classes),
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = find_class_label_from_one_hot(&vec[0], vec.size());
      break;
    case leaf_algo_t::VECTOR_LEAF: {
      ASSERT(vec.size() == static_cast<std::size_t>(forest_params.num_classes),
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = *leaf_counter;
      for (int k = 0; k < forest_params.num_classes; k++) {
        (*vector_leaf)[*leaf_counter * forest_params.num_classes + k] = vec[k];
      }
      (*leaf_counter)++;
      break;
    }
    case leaf_algo_t::FLOAT_UNARY_BINARY:
    case leaf_algo_t::GROVE_PER_CLASS:
      fil_node->val.f = static_cast<float>(tl_tree.LeafValue(tl_node_id));
      ASSERT(!tl_tree.HasLeafVector(tl_node_id),
             "some but not all treelite leaves have leaf_vector()");
      break;
    default: ASSERT(false, "internal error: invalid leaf_algo");
  };
}

template <typename fil_node_t>
struct conversion_state {
  fil_node_t node;
  bool swap_child_nodes;
};

// modifies cat_sets
template <typename fil_node_t, typename T, typename L>
conversion_state<fil_node_t> tl2fil_inner_node(int fil_left_child,
                                               const tl::Tree<T, L>& tree,
                                               int tl_node_id,
                                               cat_sets_owner* cat_sets,
                                               std::size_t* bit_pool_offset)
{
  int tl_left = tree.LeftChild(tl_node_id), tl_right = tree.RightChild(tl_node_id);
  val_t<float> split  = {.f = NAN};  // yes there's a default initializer already
  int feature_id      = tree.SplitIndex(tl_node_id);
  bool is_categorical = tree.SplitType(tl_node_id) == tl::SplitFeatureType::kCategorical &&
                        tree.MatchingCategories(tl_node_id).size() > 0;
  bool swap_child_nodes = false;
  if (tree.SplitType(tl_node_id) == tl::SplitFeatureType::kNumerical) {
    split.f = static_cast<float>(tree.Threshold(tl_node_id));
    adjust_threshold(&split.f, &swap_child_nodes, tree.ComparisonOp(tl_node_id));
  } else if (tree.SplitType(tl_node_id) == tl::SplitFeatureType::kCategorical) {
    // for FIL, the list of categories is always for the right child
    swap_child_nodes = !tree.CategoriesListRightChild(tl_node_id);
    if (tree.MatchingCategories(tl_node_id).size() > 0) {
      int sizeof_mask = cat_sets->accessor().sizeof_mask(feature_id);
      split.idx       = *bit_pool_offset;
      *bit_pool_offset += sizeof_mask;
      // cat_sets->bits have been zero-initialized
      uint8_t* bits = &cat_sets->bits[split.idx];
      for (std::uint32_t category : tree.MatchingCategories(tl_node_id)) {
        bits[category / BITS_PER_BYTE] |= 1 << (category % BITS_PER_BYTE);
      }
    } else {
      // always branch left in FIL. Already accounted for Treelite branching direction above.
      split.f = NAN;
    }
  } else {
    ASSERT(false, "only numerical and categorical split nodes are supported");
  }
  bool default_left = tree.DefaultLeft(tl_node_id) ^ swap_child_nodes;
  fil_node_t node(
    val_t<float>{}, split, feature_id, default_left, false, is_categorical, fil_left_child);
  return conversion_state<fil_node_t>{node, swap_child_nodes};
}

template <typename fil_node_t, typename T, typename L>
int tree2fil(std::vector<fil_node_t>& nodes,
             int root,
             const tl::Tree<T, L>& tree,
             std::size_t tree_idx,
             const forest_params_t& forest_params,
             std::vector<float>* vector_leaf,
             std::size_t* leaf_counter,
             cat_sets_owner* cat_sets)
{
  // needed if the node is sparse, to place within memory for the FIL tree
  int sparse_index = 1;
  walk_tree(
    tree,
    [&](int node_id, int fil_node_id) {
      // reserve space for child nodes
      // left is the offset of the left child node relative to the tree root
      // in the array of all nodes of the FIL sparse forest
      int left = node_traits<fil_node_t>::IS_DENSE ? 2 * fil_node_id + 1 : sparse_index;
      sparse_index += 2;
      conversion_state<fil_node_t> cs = tl2fil_inner_node<fil_node_t>(
        left, tree, node_id, cat_sets, &cat_sets->bit_pool_offsets[tree_idx]);
      nodes[root + fil_node_id] = cs.node;

      return cs.swap_child_nodes ? std::pair(left + 1, left) : std::pair(left, left + 1);
    },
    [&](int node_id, int fil_node_id) {
      nodes[root + fil_node_id] = fil_node_t({}, {}, 0, false, true, false, 0);
      tl2fil_leaf_payload(&nodes[root + fil_node_id],
                          root + fil_node_id,
                          tree,
                          node_id,
                          forest_params,
                          vector_leaf,
                          leaf_counter);
    });
  return root;
}

struct level_entry {
  int n_branch_nodes, n_leaves;
};
// hist has branch and leaf count given depth
template <typename T, typename L>
inline void node_depth_hist(const tl::Tree<T, L>& tree, std::vector<level_entry>& hist)
{
  walk_tree(
    tree,
    [&](int node_id, std::size_t depth) {
      if (depth >= hist.size()) hist.resize(depth + 1, {0, 0});
      hist[depth].n_branch_nodes++;
      return std::pair(depth + 1, depth + 1);
    },
    [&](int node_id, std::size_t depth) {
      if (depth >= hist.size()) hist.resize(depth + 1, {0, 0});
      hist[depth].n_leaves++;
    });
}

template <typename T, typename L>
std::stringstream depth_hist_and_max(const tl::ModelImpl<T, L>& model)
{
  using namespace std;
  vector<level_entry> hist;
  for (const auto& tree : model.trees)
    node_depth_hist(tree, hist);

  int min_leaf_depth = -1, leaves_times_depth = 0, total_branches = 0, total_leaves = 0;
  stringstream forest_shape;
  ios default_state(nullptr);
  default_state.copyfmt(forest_shape);
  forest_shape << "Depth histogram:" << endl << "depth branches leaves   nodes" << endl;
  for (std::size_t level = 0; level < hist.size(); ++level) {
    level_entry e = hist[level];
    forest_shape << setw(5) << level << setw(9) << e.n_branch_nodes << setw(7) << e.n_leaves
                 << setw(8) << e.n_branch_nodes + e.n_leaves << endl;
    forest_shape.copyfmt(default_state);
    if (e.n_leaves && min_leaf_depth == -1) min_leaf_depth = level;
    leaves_times_depth += e.n_leaves * level;
    total_branches += e.n_branch_nodes;
    total_leaves += e.n_leaves;
  }
  int total_nodes = total_branches + total_leaves;
  forest_shape << "Total: branches: " << total_branches << " leaves: " << total_leaves
               << " nodes: " << total_nodes << endl;
  forest_shape << "Avg nodes per tree: " << setprecision(2)
               << total_nodes / (float)hist[0].n_branch_nodes << endl;
  forest_shape.copyfmt(default_state);
  forest_shape << "Leaf depth: min: " << min_leaf_depth << " avg: " << setprecision(2) << fixed
               << leaves_times_depth / (float)total_leaves << " max: " << hist.size() - 1 << endl;
  forest_shape.copyfmt(default_state);

  vector<char> hist_bytes(hist.size() * sizeof(hist[0]));
  memcpy(&hist_bytes[0], &hist[0], hist_bytes.size());
  // std::hash does not promise to not be identity. Xoring plain numbers which
  // add up to one another erases information, hence, std::hash is unsuitable here
  forest_shape << "Depth histogram fingerprint: " << hex
               << fowler_noll_vo_fingerprint64_32(hist_bytes.begin(), hist_bytes.end()) << endl;
  forest_shape.copyfmt(default_state);

  return forest_shape;
}

template <typename T, typename L>
size_t tl_leaf_vector_size(const tl::ModelImpl<T, L>& model)
{
  const tl::Tree<T, L>& tree = model.trees[0];
  int node_key;
  for (node_key = tree_root(tree); !tree.IsLeaf(node_key); node_key = tree.RightChild(node_key))
    ;
  if (tree.HasLeafVector(node_key)) return tree.LeafVector(node_key).size();
  return 0;
}

// tl2fil_common is the part of conversion from a treelite model
// common for dense and sparse forests
template <typename T, typename L>
void tl2fil_common(forest_params_t* params,
                   const tl::ModelImpl<T, L>& model,
                   const treelite_params_t* tl_params)
{
  // fill in forest-indendent params
  params->algo      = tl_params->algo;
  params->threshold = tl_params->threshold;

  // fill in forest-dependent params
  params->depth = max_depth(model);  // also checks for cycles

  const tl::ModelParam& param = model.param;

  // assuming either all leaves use the .leaf_vector() or all leaves use .leaf_value()
  size_t leaf_vec_size = tl_leaf_vector_size(model);
  std::string pred_transform(param.pred_transform);
  if (leaf_vec_size > 0) {
    ASSERT(leaf_vec_size == model.task_param.num_class, "treelite model inconsistent");
    params->num_classes = leaf_vec_size;
    params->leaf_algo   = leaf_algo_t::VECTOR_LEAF;

    ASSERT(pred_transform == "max_index" || pred_transform == "identity_multiclass",
           "only max_index and identity_multiclass values of pred_transform "
           "are supported for multi-class models");

  } else {
    if (model.task_param.num_class > 1) {
      params->num_classes = static_cast<int>(model.task_param.num_class);
      ASSERT(tl_params->output_class, "output_class==true is required for multi-class models");
      ASSERT(pred_transform == "identity_multiclass" || pred_transform == "max_index" ||
               pred_transform == "softmax" || pred_transform == "multiclass_ova",
             "only identity_multiclass, max_index, multiclass_ova and softmax "
             "values of pred_transform are supported for xgboost-style "
             "multi-class classification models.");
      // this function should not know how many threads per block will be used
      params->leaf_algo = leaf_algo_t::GROVE_PER_CLASS;
    } else {
      params->num_classes = tl_params->output_class ? 2 : 1;
      ASSERT(pred_transform == "sigmoid" || pred_transform == "identity",
             "only sigmoid and identity values of pred_transform "
             "are supported for binary classification and regression models.");
      params->leaf_algo = leaf_algo_t::FLOAT_UNARY_BINARY;
    }
  }

  params->num_cols = model.num_feature;

  ASSERT(param.sigmoid_alpha == 1.0f, "sigmoid_alpha not supported");
  params->global_bias = param.global_bias;
  params->output      = output_t::RAW;
  /** output_t::CLASS denotes using a threshold in FIL, when
      predict_proba == false. For all multiclass models, the best class is
      selected using argmax instead. This happens when either
      leaf_algo == CATEGORICAL_LEAF or num_classes > 2.
  **/
  if (tl_params->output_class && params->leaf_algo != CATEGORICAL_LEAF &&
      params->num_classes <= 2) {
    params->output = output_t(params->output | output_t::CLASS);
  }
  // "random forest" in treelite means tree output averaging
  if (model.average_tree_output) { params->output = output_t(params->output | output_t::AVG); }
  if (pred_transform == "sigmoid" || pred_transform == "multiclass_ova") {
    params->output = output_t(params->output | output_t::SIGMOID);
  }
  if (pred_transform == "softmax") params->output = output_t(params->output | output_t::SOFTMAX);
  params->num_trees        = model.trees.size();
  params->blocks_per_sm    = tl_params->blocks_per_sm;
  params->threads_per_tree = tl_params->threads_per_tree;
  params->n_items          = tl_params->n_items;
}

template <typename node_t>
template <typename threshold_t, typename leaf_t>
void node_traits<node_t>::check(const treelite::ModelImpl<threshold_t, leaf_t>& model)
{
  if constexpr (!std::is_same<node_t, sparse_node8>()) return;
  const int MAX_FEATURES   = 1 << sparse_node8::FID_NUM_BITS;
  const int MAX_TREE_NODES = (1 << sparse_node8::LEFT_NUM_BITS) - 1;
  // check the number of features
  int num_features = model.num_feature;
  ASSERT(num_features <= MAX_FEATURES,
         "model has %d features, "
         "but only %d supported for 8-byte sparse nodes",
         num_features,
         MAX_FEATURES);

  // check the number of tree nodes
  const std::vector<tl::Tree<threshold_t, leaf_t>>& trees = model.trees;
  for (std::size_t i = 0; i < trees.size(); ++i) {
    int num_nodes = trees[i].num_nodes;
    ASSERT(num_nodes <= MAX_TREE_NODES,
           "tree %zu has %d nodes, "
           "but only %d supported for 8-byte sparse nodes",
           i,
           num_nodes,
           MAX_TREE_NODES);
  }
}

template <typename fil_node_t, typename threshold_t, typename leaf_t>
struct tl2fil_t {
  std::vector<int> roots_;
  std::vector<fil_node_t> nodes_;
  std::vector<float> vector_leaf_;
  forest_params_t params_;
  cat_sets_owner cat_sets_;
  const tl::ModelImpl<threshold_t, leaf_t>& model_;
  const treelite_params_t& tl_params_;

  tl2fil_t(const tl::ModelImpl<threshold_t, leaf_t>& model_, const treelite_params_t& tl_params_)
    : model_(model_), tl_params_(tl_params_)
  {
  }

  void init()
  {
    static const bool IS_DENSE = node_traits<fil_node_t>::IS_DENSE;
    tl2fil_common(&params_, model_, &tl_params_);
    node_traits<fil_node_t>::check(model_);

    std::size_t num_trees = model_.trees.size();

    std::size_t total_nodes = 0;
    roots_.reserve(num_trees);
    for (auto& tree : model_.trees) {
      roots_.push_back(total_nodes);
      total_nodes += IS_DENSE ? tree_num_nodes(params_.depth) : tree.num_nodes;
    }

    if (params_.leaf_algo == VECTOR_LEAF) {
      std::size_t max_leaves = IS_DENSE ? num_trees * (tree_num_nodes(params_.depth) + 1) / 2
                                        : (total_nodes + num_trees) / 2;
      vector_leaf_.resize(max_leaves * params_.num_classes);
    }

    cat_sets_ = allocate_cat_sets_owner(model_);
    nodes_.resize(total_nodes);

// convert the nodes_
#pragma omp parallel for
    for (std::size_t tree_idx = 0; tree_idx < num_trees; ++tree_idx) {
      // Max number of leaves processed so far
      size_t leaf_counter = (roots_[tree_idx] + tree_idx) / 2;
      tree2fil(nodes_,
               roots_[tree_idx],
               model_.trees[tree_idx],
               tree_idx,
               params_,
               &vector_leaf_,
               &leaf_counter,
               &cat_sets_);
    }

    params_.num_nodes = nodes_.size();
  }

  /// initializes FIL forest object, to be ready to infer
  void init_forest(const raft::handle_t& handle, forest_t<float>* pforest)
  {
    ML::fil::init(
      handle, pforest, cat_sets_.accessor(), vector_leaf_, roots_.data(), nodes_.data(), &params_);
    // sync is necessary as nodes_ are used in init(),
    // but destructed at the end of this function
    handle.sync_stream(handle.get_stream());
    if (tl_params_.pforest_shape_str) {
      *tl_params_.pforest_shape_str = sprintf_shape(model_, nodes_, roots_, cat_sets_);
    }
  }
};

template <typename fil_node_t, typename threshold_t, typename leaf_t>
void convert(const raft::handle_t& handle,
             forest_t<float>* pforest,
             const tl::ModelImpl<threshold_t, leaf_t>& model,
             const treelite_params_t& tl_params)
{
  tl2fil_t<fil_node_t, threshold_t, leaf_t> tl2fil(model, tl_params);
  tl2fil.init();
  tl2fil.init_forest(handle, pforest);
}

template <typename real_t>
constexpr bool type_supported()
{
  // not using std::is_floating_point because we did not instantiate fp16-based nodes/trees/forests
  return std::is_same_v<real_t, float> || std::is_same_v<real_t, double>;
}

template <typename threshold_t, typename leaf_t>
void from_treelite(const raft::handle_t& handle,
                   forest_t<float>* pforest,
                   const tl::ModelImpl<threshold_t, leaf_t>& model,
                   const treelite_params_t* tl_params)
{
  // Invariants on threshold and leaf types
  static_assert(type_supported<threshold_t>(),
                "Model must contain float32 or float64 thresholds for splits");
  ASSERT(type_supported<leaf_t>(), "Models with integer leaf output are not yet supported");
  // Display appropriate warnings when float64 values are being casted into
  // float32, as FIL only supports inferencing with float32 for the time being
  if (std::is_same<threshold_t, double>::value || std::is_same<leaf_t, double>::value) {
    CUML_LOG_WARN(
      "Casting all thresholds and leaf values to float32, as FIL currently "
      "doesn't support inferencing models with float64 values. "
      "This may lead to predictions with reduced accuracy.");
  }
  // same as std::common_type: float+double=double, float+int64_t=float
  using real_t = decltype(threshold_t(0) + leaf_t(0));

  storage_type_t storage_type = tl_params->storage_type;
  // build dense trees by default
  if (storage_type == storage_type_t::AUTO) {
    if (tl_params->algo == algo_t::ALGO_AUTO || tl_params->algo == algo_t::NAIVE) {
      int depth = max_depth(model);
      // max 2**25 dense nodes, 256 MiB dense model size. Categorical mask size is unlimited and not
      // affected by storage format.
      const int LOG2_MAX_DENSE_NODES = 25;
      int log2_num_dense_nodes       = depth + 1 + int(ceil(std::log2(model.trees.size())));
      storage_type = log2_num_dense_nodes > LOG2_MAX_DENSE_NODES ? storage_type_t::SPARSE
                                                                 : storage_type_t::DENSE;
    } else {
      // only dense storage is supported for other algorithms
      storage_type = storage_type_t::DENSE;
    }
  }

  switch (storage_type) {
    case storage_type_t::DENSE:
      convert<dense_node<float>>(handle, pforest, model, *tl_params);
      break;
    case storage_type_t::SPARSE:
      convert<sparse_node16<float>>(handle, pforest, model, *tl_params);
      break;
    case storage_type_t::SPARSE8: convert<sparse_node8>(handle, pforest, model, *tl_params); break;
    default: ASSERT(false, "tl_params->sparse must be one of AUTO, DENSE or SPARSE");
  }
}

void from_treelite(const raft::handle_t& handle,
                   forest_t<float>* pforest,
                   ModelHandle model,
                   const treelite_params_t* tl_params)
{
  const tl::Model& model_ref = *(tl::Model*)model;
  model_ref.Dispatch([&](const auto& model_inner) {
    // model_inner is of the concrete type tl::ModelImpl<threshold_t, leaf_t>
    from_treelite(handle, pforest, model_inner, tl_params);
  });
}

// allocates caller-owned char* using malloc()
template <typename threshold_t, typename leaf_t, typename node_t>
char* sprintf_shape(const tl::ModelImpl<threshold_t, leaf_t>& model,
                    const std::vector<node_t>& nodes,
                    const std::vector<int>& trees,
                    const cat_sets_owner cat_sets)
{
  std::stringstream forest_shape = depth_hist_and_max(model);
  double size_mb = (trees.size() * sizeof(trees.front()) + nodes.size() * sizeof(nodes.front()) +
                    cat_sets.bits.size()) /
                   1e6;
  forest_shape << storage_type_repr[node_traits<node_t>::storage_type_enum] << " model size "
               << std::setprecision(2) << size_mb << " MB" << std::endl;
  if (cat_sets.bits.size() > 0) {
    forest_shape << "number of categorical nodes for each feature id: {";
    std::size_t total_cat_nodes = 0;
    for (std::size_t n : cat_sets.n_nodes) {
      forest_shape << n << " ";
      total_cat_nodes += n;
    }
    forest_shape << "}" << std::endl << "total categorical nodes: " << total_cat_nodes << std::endl;
    forest_shape << "maximum matching category for each feature id: {";
    for (float fid_num_cats : cat_sets.fid_num_cats)
      forest_shape << static_cast<int>(fid_num_cats) - 1 << " ";
    forest_shape << "}" << std::endl;
  }
  // stream may be discontiguous
  std::string forest_shape_str = forest_shape.str();
  // now copy to a non-owning allocation
  char* shape_out = (char*)malloc(forest_shape_str.size() + 1);  // incl. \0
  memcpy((void*)shape_out, forest_shape_str.c_str(), forest_shape_str.size() + 1);
  return shape_out;
}

}  // namespace fil
}  // namespace ML
