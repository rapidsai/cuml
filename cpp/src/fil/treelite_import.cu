#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file treelite_import.cu converts from treelite format to a FIL-centric CPU-RAM format, so that
 * fil.cu can make a `forest` object out of it. */

#include "common.cuh"    // for num_trees, tree_num_nodes
#include "internal.cuh"  // for MAX_FIL_INT_FLOAT, BITS_PER_BYTE, cat_feature_counters, cat_sets, cat_sets_owner, categorical_sets, leaf_algo_t

#include <cuml/fil/fil.h>  // for algo_t, from_treelite, storage_type_repr, storage_type_t, treelite_params_t
#include <cuml/fil/fnv_hash.h>     // for fowler_noll_vo_fingerprint64_32
#include <cuml/common/logger.hpp>  // for CUML_LOG_WARN

#include <raft/cudart_utils.h>  // for CUDA_CHECK
#include <raft/error.hpp>       // for ASSERT
#include <raft/handle.hpp>      // for handle_t

#include <treelite/base.h>   // for Operator, SplitFeatureType, kGE, kGT, kLE, kLT, kNumerical
#include <treelite/c_api.h>  // for ModelHandle
#include <treelite/tree.h>   // for Tree

#include <omp.h>  // for omp

#include <algorithm>    // for std::max
#include <bitset>       // for std::bitset
#include <cmath>        // for NAN
#include <cstddef>      // for std::size_t
#include <cstdint>      // for uint8_t
#include <iosfwd>       // for ios, stringstream
#include <stack>        // for std::stack
#include <string>       // for std::string
#include <type_traits>  // for std::is_same

namespace ML {
namespace fil {

namespace tl = treelite;

std::ostream& operator<<(std::ostream& os, const cat_sets_owner& cso)
{
  os << "\nbits { ";
  for (uint8_t b : cso.bits) {
    os << std::bitset<BITS_PER_BYTE>(b) << " ";
  }
  os << " }\nmax_matching {";
  for (float fid_num_cats : cso.fid_num_cats) {
    os << static_cast<int>(fid_num_cats) - 1 << " ";
  }
  os << " }";
  return os;
}

template <typename T, typename L>
int tree_root(const tl::Tree<T, L>& tree)
{
  return 0;  // Treelite format assumes that the root is 0
}

template <typename T, typename L>
inline int max_depth(const tl::Tree<T, L>& tree)
{
  // trees of this depth aren't used, so it most likely means bad input data,
  // e.g. cycles in the forest
  const int DEPTH_LIMIT = 500;
  int root_index        = tree_root(tree);
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  stack.push(pair_t(root_index, 0));
  int max_depth = 0;
  while (!stack.empty()) {
    const pair_t& pair = stack.top();
    int node_id        = pair.first;
    int depth          = pair.second;
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      stack.push(pair_t(tree.LeftChild(node_id), depth + 1));
      node_id = tree.RightChild(node_id);
      depth++;
      ASSERT(depth < DEPTH_LIMIT, "depth limit reached, might be a cycle in the tree");
    }
    // only need to update depth for leaves
    max_depth = std::max(max_depth, depth);
  }
  return max_depth;
}

template <typename T, typename L>
int max_depth(const tl::ModelImpl<T, L>& model)
{
  int depth         = 0;
  const auto& trees = model.trees;
#pragma omp parallel for reduction(max : depth)
  for (size_t i = 0; i < trees.size(); ++i) {
    const auto& tree = trees[i];
    depth            = std::max(depth, max_depth(tree));
  }
  return depth;
}

void elementwise_combine(std::vector<cat_feature_counters>& dst,
                         const std::vector<cat_feature_counters>& extra)
{
  std::transform(dst.begin(), dst.end(), extra.begin(), dst.begin(), cat_feature_counters::combine);
}

// constructs a vector of size n_cols (number of features, or columns) from a Treelite tree,
// where each feature has a maximum matching category and node count (from this tree alone).
template <typename T, typename L>
inline std::vector<cat_feature_counters> cat_counter_vec(const tl::Tree<T, L>& tree, int n_cols)
{
  std::vector<cat_feature_counters> res(n_cols);
  std::stack<int> stack;
  stack.push(tree_root(tree));
  while (!stack.empty()) {
    int node_id = stack.top();
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      if (tree.SplitType(node_id) == tl::SplitFeatureType::kCategorical) {
        std::vector<std::uint32_t> mmv = tree.MatchingCategories(node_id);
        int max_matching_cat;
        if (mmv.size() > 0) {
          // in `struct cat_feature_counters` and GPU structures, max matching category is an int
          // cast is safe because all precise int floats fit into ints, which are asserted to be 32
          // bits
          max_matching_cat = mmv.back();
          ASSERT(max_matching_cat <= MAX_FIL_INT_FLOAT,
                 "FIL cannot infer on "
                 "more than %d matching categories",
                 MAX_FIL_INT_FLOAT);
        } else {
          max_matching_cat = -1;
        }
        cat_feature_counters& counters = res[tree.SplitIndex(node_id)];
        counters =
          cat_feature_counters::combine(counters, cat_feature_counters{max_matching_cat, 1});
      }
      stack.push(tree.LeftChild(node_id));
      node_id = tree.RightChild(node_id);
    }
  }
  return res;
}

// computes overall categorical bit pool size for a tree imported from the Treelite tree
template <typename T, typename L>
inline std::size_t bit_pool_size(const tl::Tree<T, L>& tree, const categorical_sets& cat_sets)
{
  std::size_t size = 0;
  std::stack<int> stack;
  stack.push(tree_root(tree));
  while (!stack.empty()) {
    int node_id = stack.top();
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      if (tree.SplitType(node_id) == tl::SplitFeatureType::kCategorical &&
          tree.MatchingCategories(node_id).size() > 0) {
        int fid = tree.SplitIndex(node_id);
        size += cat_sets.sizeof_mask(fid);
      }
      stack.push(tree.LeftChild(node_id));
      node_id = tree.RightChild(node_id);
    }
  }
  return size;
}

template <typename T, typename L>
cat_sets_owner allocate_cat_sets_owner(const tl::ModelImpl<T, L>& model)
{
#pragma omp declare reduction(cat_counter_vec_red : std::vector<cat_feature_counters> \
      : elementwise_combine(omp_out, omp_in))                 \
    initializer(omp_priv = omp_orig)
  const auto& trees = model.trees;
  cat_sets_owner cat_sets;
  std::vector<cat_feature_counters> counters(model.num_feature);
#pragma omp parallel for reduction(cat_counter_vec_red : counters)
  for (std::size_t i = 0; i < trees.size(); ++i) {
    elementwise_combine(counters, cat_counter_vec(trees[i], model.num_feature));
  }
  cat_sets.consume_counters(counters);
  std::vector<std::size_t> bit_pool_sizes(trees.size());
#pragma omp parallel for
  for (std::size_t i = 0; i < trees.size(); ++i) {
    bit_pool_sizes[i] = bit_pool_size(trees[i], cat_sets.accessor());
  }
  cat_sets.consume_bit_pool_sizes(bit_pool_sizes);
  return cat_sets;
}

void adjust_threshold(
  float* pthreshold, int* tl_left, int* tl_right, bool* default_left, tl::Operator comparison_op)
{
  // in treelite (take left node if val [op] threshold),
  // the meaning of the condition is reversed compared to FIL;
  // thus, "<" in treelite corresonds to comparison ">=" used by FIL
  // https://github.com/dmlc/treelite/blob/master/include/treelite/tree.h#L243
  if (isnan(*pthreshold)) {
    std::swap(*tl_left, *tl_right);
    *default_left = !*default_left;
    return;
  }
  switch (comparison_op) {
    case tl::Operator::kLT: break;
    case tl::Operator::kLE:
      // x <= y is equivalent to x < y', where y' is the next representable float
      *pthreshold = std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
      break;
    case tl::Operator::kGT:
      // x > y is equivalent to x >= y', where y' is the next representable float
      // left and right still need to be swapped
      *pthreshold = std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
    case tl::Operator::kGE:
      // swap left and right
      std::swap(*tl_left, *tl_right);
      *default_left = !*default_left;
      break;
    default: ASSERT(false, "only <, >, <= and >= comparisons are supported");
  }
}

/** if the vector consists of zeros and a single one, return the position
for the one (assumed class label). Else, asserts false.
If the vector contains a NAN, asserts false */
template <typename L>
int find_class_label_from_one_hot(L* vector, int len)
{
  bool found_label = false;
  int out;
  for (int i = 0; i < len; ++i) {
    if (vector[i] == static_cast<L>(1.0)) {
      ASSERT(!found_label, "label vector contains multiple 1.0f");
      out         = i;
      found_label = true;
    } else {
      ASSERT(vector[i] == static_cast<L>(0.0),
             "label vector contains values other than 0.0 and 1.0");
    }
  }
  ASSERT(found_label, "did not find 1.0f in vector");
  return out;
}

template <typename fil_node_t, typename T, typename L>
void tl2fil_leaf_payload(fil_node_t* fil_node,
                         int fil_node_id,
                         const tl::Tree<T, L>& tl_tree,
                         int tl_node_id,
                         const forest_params_t& forest_params,
                         std::vector<float>* vector_leaf,
                         size_t* leaf_counter)
{
  auto vec = tl_tree.LeafVector(tl_node_id);
  switch (forest_params.leaf_algo) {
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(vec.size() == static_cast<std::size_t>(forest_params.num_classes),
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = find_class_label_from_one_hot(&vec[0], vec.size());
      break;
    case leaf_algo_t::VECTOR_LEAF: {
      ASSERT(vec.size() == static_cast<std::size_t>(forest_params.num_classes),
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = *leaf_counter;
      for (int k = 0; k < forest_params.num_classes; k++) {
        (*vector_leaf)[*leaf_counter * forest_params.num_classes + k] = vec[k];
      }
      (*leaf_counter)++;
      break;
    }
    case leaf_algo_t::FLOAT_UNARY_BINARY:
    case leaf_algo_t::GROVE_PER_CLASS:
      fil_node->val.f = static_cast<float>(tl_tree.LeafValue(tl_node_id));
      ASSERT(!tl_tree.HasLeafVector(tl_node_id),
             "some but not all treelite leaves have leaf_vector()");
      break;
    default: ASSERT(false, "internal error: invalid leaf_algo");
  };
}

template <typename fil_node_t>
struct conversion_state {
  fil_node_t node;
  int tl_left;
  int tl_right;
};

// modifies cat_sets
template <typename fil_node_t, typename T, typename L>
conversion_state<fil_node_t> tl2fil_inner_node(int fil_left_child,
                                               const tl::Tree<T, L>& tree,
                                               int tl_node_id,
                                               const forest_params_t& forest_params,
                                               cat_sets_owner* cat_sets,
                                               std::size_t* bit_pool_offset)
{
  int tl_left = tree.LeftChild(tl_node_id), tl_right = tree.RightChild(tl_node_id);
  val_t split         = {.f = NAN};  // yes there's a default initializer already
  int feature_id      = tree.SplitIndex(tl_node_id);
  bool is_categorical = tree.SplitType(tl_node_id) == tl::SplitFeatureType::kCategorical &&
                        tree.MatchingCategories(tl_node_id).size() > 0;
  bool default_left = tree.DefaultLeft(tl_node_id);
  if (tree.SplitType(tl_node_id) == tl::SplitFeatureType::kNumerical) {
    split.f = static_cast<float>(tree.Threshold(tl_node_id));
    adjust_threshold(&split.f, &tl_left, &tl_right, &default_left, tree.ComparisonOp(tl_node_id));
  } else if (tree.SplitType(tl_node_id) == tl::SplitFeatureType::kCategorical) {
    // for FIL, the list of categories is always for the right child
    if (!tree.CategoriesListRightChild(tl_node_id)) {
      std::swap(tl_left, tl_right);
      default_left = !default_left;
    }
    if (tree.MatchingCategories(tl_node_id).size() > 0) {
      int sizeof_mask = cat_sets->accessor().sizeof_mask(feature_id);
      split.idx       = *bit_pool_offset;
      *bit_pool_offset += sizeof_mask;
      // cat_sets->bits have been zero-initialized
      uint8_t* bits = &cat_sets->bits[split.idx];
      for (std::uint32_t category : tree.MatchingCategories(tl_node_id)) {
        bits[category / BITS_PER_BYTE] |= 1 << (category % BITS_PER_BYTE);
      }
    } else {
      // always branch left in FIL. Already accounted for Treelite branching direction above.
      split.f = NAN;
    }
  } else {
    ASSERT(false, "only numerical and categorical split nodes are supported");
  }
  fil_node_t node;
  if constexpr (std::is_same<fil_node_t, dense_node>()) {
    node = fil_node_t({}, split, feature_id, default_left, false, is_categorical);
  } else {
    node = fil_node_t({}, split, feature_id, default_left, false, is_categorical, fil_left_child);
  }
  return conversion_state<fil_node_t>{node, tl_left, tl_right};
}

template <typename T, typename L>
void node2fil_dense(std::vector<dense_node>* pnodes,
                    int root,
                    int cur,
                    const tl::Tree<T, L>& tree,
                    int node_id,
                    const forest_params_t& forest_params,
                    std::vector<float>* vector_leaf,
                    std::size_t* leaf_counter,
                    cat_sets_owner* cat_sets,
                    std::size_t* bit_pool_offset)
{
  if (tree.IsLeaf(node_id)) {
    (*pnodes)[root + cur] = dense_node({}, {}, 0, false, true, false);
    tl2fil_leaf_payload(
      &(*pnodes)[root + cur], root + cur, tree, node_id, forest_params, vector_leaf, leaf_counter);
    return;
  }

  // inner node
  int left = 2 * cur + 1;
  conversion_state<dense_node> cs =
    tl2fil_inner_node<dense_node>(left, tree, node_id, forest_params, cat_sets, bit_pool_offset);
  (*pnodes)[root + cur] = cs.node;
  node2fil_dense(pnodes,
                 root,
                 left,
                 tree,
                 cs.tl_left,
                 forest_params,
                 vector_leaf,
                 leaf_counter,
                 cat_sets,
                 bit_pool_offset);
  node2fil_dense(pnodes,
                 root,
                 left + 1,
                 tree,
                 cs.tl_right,
                 forest_params,
                 vector_leaf,
                 leaf_counter,
                 cat_sets,
                 bit_pool_offset);
}

template <typename T, typename L>
void tree2fil_dense(std::vector<dense_node>* pnodes,
                    int root,
                    const tl::Tree<T, L>& tree,
                    std::size_t tree_idx,
                    const forest_params_t& forest_params,
                    std::vector<float>* vector_leaf,
                    std::size_t* leaf_counter,
                    cat_sets_owner* cat_sets)
{
  node2fil_dense(pnodes,
                 root,
                 0,
                 tree,
                 tree_root(tree),
                 forest_params,
                 vector_leaf,
                 leaf_counter,
                 cat_sets,
                 &cat_sets->bit_pool_offsets[tree_idx]);
}

template <typename fil_node_t, typename T, typename L>
int tree2fil_sparse(std::vector<fil_node_t>& nodes,
                    int root,
                    const tl::Tree<T, L>& tree,
                    std::size_t tree_idx,
                    const forest_params_t& forest_params,
                    std::vector<float>* vector_leaf,
                    std::size_t* leaf_counter,
                    cat_sets_owner* cat_sets)
{
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  int built_index = root + 1;
  stack.push(pair_t(tree_root(tree), 0));
  while (!stack.empty()) {
    const pair_t& top = stack.top();
    int node_id       = top.first;
    int cur           = top.second;
    stack.pop();

    while (!tree.IsLeaf(node_id)) {
      // reserve space for child nodes
      // left is the offset of the left child node relative to the tree root
      // in the array of all nodes of the FIL sparse forest
      int left = built_index - root;
      built_index += 2;
      conversion_state<fil_node_t> cs = tl2fil_inner_node<fil_node_t>(
        left, tree, node_id, forest_params, cat_sets, &cat_sets->bit_pool_offsets[tree_idx]);
      nodes[root + cur] = cs.node;
      // push child nodes into the stack
      stack.push(pair_t(cs.tl_right, left + 1));
      // stack.push(pair_t(tl_left, left));
      node_id = cs.tl_left;
      cur     = left;
    }

    // leaf node
    nodes[root + cur] = fil_node_t({}, {}, 0, false, true, false, 0);
    tl2fil_leaf_payload(
      &nodes[root + cur], root + cur, tree, node_id, forest_params, vector_leaf, leaf_counter);
  }

  return root;
}

struct level_entry {
  int n_branch_nodes, n_leaves;
};
typedef std::pair<int, int> pair_t;
// hist has branch and leaf count given depth
template <typename T, typename L>
inline void tree_depth_hist(const tl::Tree<T, L>& tree, std::vector<level_entry>& hist)
{
  std::stack<pair_t> stack;  // {tl_id, depth}
  stack.push({tree_root(tree), 0});
  while (!stack.empty()) {
    const pair_t& top = stack.top();
    int node_id       = top.first;
    int depth         = top.second;
    stack.pop();

    while (!tree.IsLeaf(node_id)) {
      if (static_cast<std::size_t>(depth) >= hist.size()) hist.resize(depth + 1, {0, 0});
      hist[depth].n_branch_nodes++;
      stack.push({tree.LeftChild(node_id), depth + 1});
      node_id = tree.RightChild(node_id);
      depth++;
    }

    if (static_cast<std::size_t>(depth) >= hist.size()) hist.resize(depth + 1, {0, 0});
    hist[depth].n_leaves++;
  }
}

template <typename T, typename L>
std::stringstream depth_hist_and_max(const tl::ModelImpl<T, L>& model)
{
  using namespace std;
  vector<level_entry> hist;
  for (const auto& tree : model.trees)
    tree_depth_hist(tree, hist);

  int min_leaf_depth = -1, leaves_times_depth = 0, total_branches = 0, total_leaves = 0;
  stringstream forest_shape;
  ios default_state(nullptr);
  default_state.copyfmt(forest_shape);
  forest_shape << "Depth histogram:" << endl << "depth branches leaves   nodes" << endl;
  for (std::size_t level = 0; level < hist.size(); ++level) {
    level_entry e = hist[level];
    forest_shape << setw(5) << level << setw(9) << e.n_branch_nodes << setw(7) << e.n_leaves
                 << setw(8) << e.n_branch_nodes + e.n_leaves << endl;
    forest_shape.copyfmt(default_state);
    if (e.n_leaves && min_leaf_depth == -1) min_leaf_depth = level;
    leaves_times_depth += e.n_leaves * level;
    total_branches += e.n_branch_nodes;
    total_leaves += e.n_leaves;
  }
  int total_nodes = total_branches + total_leaves;
  forest_shape << "Total: branches: " << total_branches << " leaves: " << total_leaves
               << " nodes: " << total_nodes << endl;
  forest_shape << "Avg nodes per tree: " << setprecision(2)
               << total_nodes / (float)hist[0].n_branch_nodes << endl;
  forest_shape.copyfmt(default_state);
  forest_shape << "Leaf depth: min: " << min_leaf_depth << " avg: " << setprecision(2) << fixed
               << leaves_times_depth / (float)total_leaves << " max: " << hist.size() - 1 << endl;
  forest_shape.copyfmt(default_state);

  vector<char> hist_bytes(hist.size() * sizeof(hist[0]));
  memcpy(&hist_bytes[0], &hist[0], hist_bytes.size());
  // std::hash does not promise to not be identity. Xoring plain numbers which
  // add up to one another erases information, hence, std::hash is unsuitable here
  forest_shape << "Depth histogram fingerprint: " << hex
               << fowler_noll_vo_fingerprint64_32(hist_bytes.begin(), hist_bytes.end()) << endl;
  forest_shape.copyfmt(default_state);

  return forest_shape;
}

template <typename T, typename L>
size_t tl_leaf_vector_size(const tl::ModelImpl<T, L>& model)
{
  const tl::Tree<T, L>& tree = model.trees[0];
  int node_key;
  for (node_key = tree_root(tree); !tree.IsLeaf(node_key); node_key = tree.RightChild(node_key))
    ;
  if (tree.HasLeafVector(node_key)) return tree.LeafVector(node_key).size();
  return 0;
}

// tl2fil_common is the part of conversion from a treelite model
// common for dense and sparse forests
template <typename T, typename L>
void tl2fil_common(forest_params_t* params,
                   const tl::ModelImpl<T, L>& model,
                   const treelite_params_t* tl_params)
{
  // fill in forest-indendent params
  params->algo      = tl_params->algo;
  params->threshold = tl_params->threshold;

  // fill in forest-dependent params
  params->depth = max_depth(model);  // also checks for cycles

  const tl::ModelParam& param = model.param;

  // assuming either all leaves use the .leaf_vector() or all leaves use .leaf_value()
  size_t leaf_vec_size = tl_leaf_vector_size(model);
  std::string pred_transform(param.pred_transform);
  if (leaf_vec_size > 0) {
    ASSERT(leaf_vec_size == model.task_param.num_class, "treelite model inconsistent");
    params->num_classes = leaf_vec_size;
    params->leaf_algo   = leaf_algo_t::VECTOR_LEAF;

    ASSERT(pred_transform == "max_index" || pred_transform == "identity_multiclass",
           "only max_index and identity_multiclass values of pred_transform "
           "are supported for multi-class models");

  } else {
    if (model.task_param.num_class > 1) {
      params->num_classes = static_cast<int>(model.task_param.num_class);
      ASSERT(tl_params->output_class, "output_class==true is required for multi-class models");
      ASSERT(pred_transform == "identity_multiclass" || pred_transform == "max_index" ||
               pred_transform == "softmax" || pred_transform == "multiclass_ova",
             "only identity_multiclass, max_index, multiclass_ova and softmax "
             "values of pred_transform are supported for xgboost-style "
             "multi-class classification models.");
      // this function should not know how many threads per block will be used
      params->leaf_algo = leaf_algo_t::GROVE_PER_CLASS;
    } else {
      params->num_classes = tl_params->output_class ? 2 : 1;
      ASSERT(pred_transform == "sigmoid" || pred_transform == "identity",
             "only sigmoid and identity values of pred_transform "
             "are supported for binary classification and regression models.");
      params->leaf_algo = leaf_algo_t::FLOAT_UNARY_BINARY;
    }
  }

  params->num_cols = model.num_feature;

  ASSERT(param.sigmoid_alpha == 1.0f, "sigmoid_alpha not supported");
  params->global_bias = param.global_bias;
  params->output      = output_t::RAW;
  /** output_t::CLASS denotes using a threshold in FIL, when
      predict_proba == false. For all multiclass models, the best class is
      selected using argmax instead. This happens when either
      leaf_algo == CATEGORICAL_LEAF or num_classes > 2.
  **/
  if (tl_params->output_class && params->leaf_algo != CATEGORICAL_LEAF &&
      params->num_classes <= 2) {
    params->output = output_t(params->output | output_t::CLASS);
  }
  // "random forest" in treelite means tree output averaging
  if (model.average_tree_output) { params->output = output_t(params->output | output_t::AVG); }
  if (pred_transform == "sigmoid" || pred_transform == "multiclass_ova") {
    params->output = output_t(params->output | output_t::SIGMOID);
  }
  if (pred_transform == "softmax") params->output = output_t(params->output | output_t::SOFTMAX);
  params->num_trees        = model.trees.size();
  params->blocks_per_sm    = tl_params->blocks_per_sm;
  params->threads_per_tree = tl_params->threads_per_tree;
  params->n_items          = tl_params->n_items;
}

// uses treelite model with additional tl_params to initialize FIL params
// and dense nodes (stored in *pnodes)
template <typename threshold_t, typename leaf_t>
void tl2fil_dense(std::vector<dense_node>* pnodes,
                  forest_params_t* params,
                  const tl::ModelImpl<threshold_t, leaf_t>& model,
                  const treelite_params_t* tl_params,
                  cat_sets_owner* cat_sets,
                  std::vector<float>* vector_leaf)
{
  tl2fil_common(params, model, tl_params);

  // convert the nodes
  int num_nodes           = forest_num_nodes(params->num_trees, params->depth);
  int max_leaves_per_tree = (tree_num_nodes(params->depth) + 1) / 2;
  if (params->leaf_algo == VECTOR_LEAF) {
    vector_leaf->resize(max_leaves_per_tree * params->num_trees * params->num_classes);
  }
  *cat_sets = allocate_cat_sets_owner(model);
  pnodes->resize(num_nodes, dense_node());
  for (std::size_t i = 0; i < model.trees.size(); ++i) {
    size_t leaf_counter = max_leaves_per_tree * i;
    tree2fil_dense(pnodes,
                   i * tree_num_nodes(params->depth),
                   model.trees[i],
                   i,
                   *params,
                   vector_leaf,
                   &leaf_counter,
                   cat_sets);
  }
}

template <typename fil_node_t>
struct tl2fil_sparse_check_t {
  template <typename threshold_t, typename leaf_t>
  static void check(const tl::ModelImpl<threshold_t, leaf_t>& model)
  {
    ASSERT(false,
           "internal error: "
           "only a specialization of this template should be used");
  }
};

template <>
struct tl2fil_sparse_check_t<sparse_node16> {
  // no extra check for 16-byte sparse nodes
  template <typename threshold_t, typename leaf_t>
  static void check(const tl::ModelImpl<threshold_t, leaf_t>& model)
  {
  }
};

template <>
struct tl2fil_sparse_check_t<sparse_node8> {
  static const int MAX_FEATURES   = 1 << sparse_node8::FID_NUM_BITS;
  static const int MAX_TREE_NODES = (1 << sparse_node8::LEFT_NUM_BITS) - 1;
  template <typename threshold_t, typename leaf_t>
  static void check(const tl::ModelImpl<threshold_t, leaf_t>& model)
  {
    // check the number of features
    int num_features = model.num_feature;
    ASSERT(num_features <= MAX_FEATURES,
           "model has %d features, "
           "but only %d supported for 8-byte sparse nodes",
           num_features,
           MAX_FEATURES);

    // check the number of tree nodes
    const std::vector<tl::Tree<threshold_t, leaf_t>>& trees = model.trees;
    for (std::size_t i = 0; i < trees.size(); ++i) {
      int num_nodes = trees[i].num_nodes;
      ASSERT(num_nodes <= MAX_TREE_NODES,
             "tree %zu has %d nodes, "
             "but only %d supported for 8-byte sparse nodes",
             i,
             num_nodes,
             MAX_TREE_NODES);
    }
  }
};

// uses treelite model with additional tl_params to initialize FIL params,
// trees (stored in *ptrees) and sparse nodes (stored in *pnodes)
template <typename fil_node_t, typename threshold_t, typename leaf_t>
void tl2fil_sparse(std::vector<int>* ptrees,
                   std::vector<fil_node_t>* pnodes,
                   forest_params_t* params,
                   const tl::ModelImpl<threshold_t, leaf_t>& model,
                   const treelite_params_t* tl_params,
                   cat_sets_owner* cat_sets,
                   std::vector<float>* vector_leaf)
{
  tl2fil_common(params, model, tl_params);
  tl2fil_sparse_check_t<fil_node_t>::check(model);

  size_t num_trees = model.trees.size();

  ptrees->reserve(num_trees);
  ptrees->push_back(0);
  for (size_t i = 0; i < num_trees - 1; ++i) {
    ptrees->push_back(model.trees[i].num_nodes + ptrees->back());
  }
  size_t total_nodes = ptrees->back() + model.trees.back().num_nodes;

  if (params->leaf_algo == VECTOR_LEAF) {
    size_t max_leaves = (total_nodes + num_trees) / 2;
    vector_leaf->resize(max_leaves * params->num_classes);
  }

  *cat_sets = allocate_cat_sets_owner(model);
  pnodes->resize(total_nodes);

// convert the nodes
#pragma omp parallel for
  for (std::size_t i = 0; i < num_trees; ++i) {
    // Max number of leaves processed so far
    size_t leaf_counter = ((*ptrees)[i] + i) / 2;
    tree2fil_sparse(
      *pnodes, (*ptrees)[i], model.trees[i], i, *params, vector_leaf, &leaf_counter, cat_sets);
  }

  params->num_nodes = pnodes->size();
}

template <typename threshold_t, typename leaf_t>
void from_treelite(const raft::handle_t& handle,
                   forest_t* pforest,
                   const tl::ModelImpl<threshold_t, leaf_t>& model,
                   const treelite_params_t* tl_params)
{
  // Invariants on threshold and leaf types
  static_assert(std::is_same<threshold_t, float>::value || std::is_same<threshold_t, double>::value,
                "Model must contain float32 or float64 thresholds for splits");
  ASSERT((std::is_same<leaf_t, float>::value || std::is_same<leaf_t, double>::value),
         "Models with integer leaf output are not yet supported");
  // Display appropriate warnings when float64 values are being casted into
  // float32, as FIL only supports inferencing with float32 for the time being
  if (std::is_same<threshold_t, double>::value || std::is_same<leaf_t, double>::value) {
    CUML_LOG_WARN(
      "Casting all thresholds and leaf values to float32, as FIL currently "
      "doesn't support inferencing models with float64 values. "
      "This may lead to predictions with reduced accuracy.");
  }

  storage_type_t storage_type = tl_params->storage_type;
  // build dense trees by default
  if (storage_type == storage_type_t::AUTO) {
    if (tl_params->algo == algo_t::ALGO_AUTO || tl_params->algo == algo_t::NAIVE) {
      int depth = max_depth(model);
      // max 2**25 dense nodes, 256 MiB dense model size. Categorical mask size is unlimited and not
      // affected by storage format.
      const int LOG2_MAX_DENSE_NODES = 25;
      int log2_num_dense_nodes       = depth + 1 + int(ceil(std::log2(model.trees.size())));
      storage_type = log2_num_dense_nodes > LOG2_MAX_DENSE_NODES ? storage_type_t::SPARSE
                                                                 : storage_type_t::DENSE;
    } else {
      // only dense storage is supported for other algorithms
      storage_type = storage_type_t::DENSE;
    }
  }

  forest_params_t params;
  cat_sets_owner cat_sets;
  switch (storage_type) {
    case storage_type_t::DENSE: {
      std::vector<dense_node> nodes;
      std::vector<float> vector_leaf;
      tl2fil_dense(&nodes, &params, model, tl_params, &cat_sets, &vector_leaf);
      init_dense(handle, pforest, cat_sets.accessor(), vector_leaf, nodes.data(), &params);
      // sync is necessary as nodes is used in init_dense(),
      // but destructed at the end of this function
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      if (tl_params->pforest_shape_str) {
        *tl_params->pforest_shape_str = sprintf_shape(model, storage_type, nodes, {}, cat_sets);
      }
      break;
    }
    case storage_type_t::SPARSE: {
      std::vector<int> trees;
      std::vector<sparse_node16> nodes;
      std::vector<float> vector_leaf;
      tl2fil_sparse(&trees, &nodes, &params, model, tl_params, &cat_sets, &vector_leaf);
      init_sparse(
        handle, pforest, cat_sets.accessor(), vector_leaf, trees.data(), nodes.data(), &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      if (tl_params->pforest_shape_str) {
        *tl_params->pforest_shape_str = sprintf_shape(model, storage_type, nodes, trees, cat_sets);
      }
      break;
    }
    case storage_type_t::SPARSE8: {
      std::vector<int> trees;
      std::vector<sparse_node8> nodes;
      std::vector<float> vector_leaf;
      tl2fil_sparse(&trees, &nodes, &params, model, tl_params, &cat_sets, &vector_leaf);
      init_sparse(
        handle, pforest, cat_sets.accessor(), vector_leaf, trees.data(), nodes.data(), &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      if (tl_params->pforest_shape_str) {
        *tl_params->pforest_shape_str = sprintf_shape(model, storage_type, nodes, trees, cat_sets);
      }
      break;
    }
    default: ASSERT(false, "tl_params->sparse must be one of AUTO, DENSE or SPARSE");
  }
}

void from_treelite(const raft::handle_t& handle,
                   forest_t* pforest,
                   ModelHandle model,
                   const treelite_params_t* tl_params)
{
  const tl::Model& model_ref = *(tl::Model*)model;
  model_ref.Dispatch([&](const auto& model_inner) {
    // model_inner is of the concrete type tl::ModelImpl<threshold_t, leaf_t>
    from_treelite(handle, pforest, model_inner, tl_params);
  });
}

// allocates caller-owned char* using malloc()
template <typename threshold_t, typename leaf_t, typename node_t>
char* sprintf_shape(const tl::ModelImpl<threshold_t, leaf_t>& model,
                    storage_type_t storage,
                    const std::vector<node_t>& nodes,
                    const std::vector<int>& trees,
                    const cat_sets_owner cat_sets)
{
  std::stringstream forest_shape = depth_hist_and_max(model);
  double size_mb = (trees.size() * sizeof(trees.front()) + nodes.size() * sizeof(nodes.front()) +
                    cat_sets.bits.size()) /
                   1e6;
  forest_shape << storage_type_repr[storage] << " model size " << std::setprecision(2) << size_mb
               << " MB" << std::endl;
  if (cat_sets.bits.size() > 0) {
    forest_shape << "number of categorical nodes for each feature id: {";
    std::size_t total_cat_nodes = 0;
    for (std::size_t n : cat_sets.n_nodes) {
      forest_shape << n << " ";
      total_cat_nodes += n;
    }
    forest_shape << "}" << std::endl << "total categorical nodes: " << total_cat_nodes << std::endl;
    forest_shape << "maximum matching category for each feature id: {";
    for (float fid_num_cats : cat_sets.fid_num_cats)
      forest_shape << static_cast<int>(fid_num_cats) - 1 << " ";
    forest_shape << "}" << std::endl;
  }
  // stream may be discontiguous
  std::string forest_shape_str = forest_shape.str();
  // now copy to a non-owning allocation
  char* shape_out = (char*)malloc(forest_shape_str.size() + 1);  // incl. \0
  memcpy((void*)shape_out, forest_shape_str.c_str(), forest_shape_str.size() + 1);
  return shape_out;
}

}  // namespace fil
}  // namespace ML
