#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common.cuh"

#include "internal.cuh"

#include <cuml/fil/multi_sum.cuh>

#include <raft/cuda_utils.cuh>
#include <raft/cudart_utils.h>

#include <thrust/functional.h>

#include <algorithm>
#include <cmath>

#ifndef CUDA_PRAGMA_UNROLL
#ifdef __CUDA_ARCH__
#define CUDA_PRAGMA_UNROLL _Pragma("unroll")
#else
#define CUDA_PRAGMA_UNROLL
#endif  // __CUDA_ARCH__
#endif  // CUDA_PRAGMA_UNROLL

#define INLINE_CONFIG __forceinline__

namespace ML {
namespace fil {

// vec wraps float[N], int[N] or double[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec;

template <typename BinaryOp>
struct Vectorized {
  BinaryOp op;
  __host__ __device__ Vectorized(BinaryOp op_) : op(op_) {}
  template <int NITEMS, typename T>
  constexpr __host__ __device__ __forceinline__ vec<NITEMS, T> operator()(vec<NITEMS, T> a,
                                                                          vec<NITEMS, T> b) const
  {
    vec<NITEMS, T> c;
    CUDA_PRAGMA_UNROLL
    for (int i = 0; i < NITEMS; i++)
      c[i] = op(a[i], b[i]);
    return c;
  }
};
template <typename BinaryOp>
constexpr __host__ __device__ Vectorized<BinaryOp> vectorized(BinaryOp op)
{
  return Vectorized<BinaryOp>(op);
}

template <int N, typename T>
struct vec {
  static const int NITEMS = N;
  T data[N];
  explicit __host__ __device__ vec(T t)
  {
    CUDA_PRAGMA_UNROLL
    for (int i = 0; i < N; ++i)
      data[i] = t;
  }
  __host__ __device__ vec() : vec(T()) {}
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a, const vec<N, T>& b)
  {
    return vectorized(hipcub::Sum())(a, b);
  }
  friend __host__ __device__ void operator+=(vec<N, T>& a, const vec<N, T>& b) { a = a + b; }
  template <typename Vec>
  friend __host__ __device__ vec<N, T> operator/(vec<N, T>& a, const Vec& b)
  {
    return vectorized(thrust::divides<T>())(a, vec<N, T>(b));
  }
  template <typename Vec>
  friend __host__ __device__ void operator/=(vec<N, T>& a, const Vec& b)
  {
    a = a / b;
  }
};

template <typename real_t>
struct best_margin_label : hipcub::KeyValuePair<int, real_t> {
  __host__ __device__ best_margin_label(hipcub::KeyValuePair<int, real_t> pair)
    : hipcub::KeyValuePair<int, real_t>(pair)
  {
  }
  __host__ __device__ best_margin_label(int c = 0, real_t f = -INFINITY)
    : hipcub::KeyValuePair<int, real_t>({c, f})
  {
  }
};

template <int NITEMS, typename real_t>
__device__ __forceinline__ vec<NITEMS, best_margin_label<real_t>> to_vec(int c,
                                                                         vec<NITEMS, real_t> margin)
{
  vec<NITEMS, best_margin_label<real_t>> ret;
  CUDA_PRAGMA_UNROLL
  for (int i = 0; i < NITEMS; ++i)
    ret[i] = best_margin_label<real_t>(c, margin[i]);
  return ret;
}

struct ArgMax {
  template <int NITEMS, typename real_t>
  __host__ __device__ __forceinline__ vec<NITEMS, best_margin_label<real_t>> operator()(
    vec<NITEMS, best_margin_label<real_t>> a, vec<NITEMS, best_margin_label<real_t>> b) const
  {
    vec<NITEMS, best_margin_label<real_t>> c;
    CUDA_PRAGMA_UNROLL
    for (int i = 0; i < NITEMS; i++)
      c[i] = hipcub::ArgMax()(a[i], b[i]);
    return c;
  }
};

/** tree_leaf_output returns the leaf outputs from the tree with leaf indices
    given by leaves for n_rows items. FULL_ITEMS indicates whether n_rows ==
    NITEMS, to allow the compiler to skip the conditional when unrolling the
    loop. */
template <typename output_type, bool FULL_NITEMS, int NITEMS, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> tree_leaf_output(tree_type tree,
                                                                     int n_rows,
                                                                     int (&leaves)[NITEMS])
{
  vec<NITEMS, output_type> out(0);
  CUDA_PRAGMA_UNROLL
  for (int j = 0; j < NITEMS; ++j) {
    if (FULL_NITEMS || j < n_rows) {
      /** dependent names are not considered templates by default, unless it's a
          member of a current [template] instantiation. As output<>() is a
          member function inherited from the base class, template
          output<output_type>() is required. */
      out[j] = tree[leaves[j]].template output<output_type>();
    }
  }
  return out;
}

template <int NITEMS, bool CATS_SUPPORTED, typename output_type, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> infer_one_tree(
  tree_type tree, const typename tree_type::real_type* input, int cols, int n_rows)
{
  // find the leaf nodes for each row
  int curr[NITEMS];
  // the first n_rows are active
  int mask = (1 << n_rows) - 1;
  for (int j = 0; j < NITEMS; ++j)
    curr[j] = 0;
  do {
    CUDA_PRAGMA_UNROLL
    for (int j = 0; j < NITEMS; ++j) {
      auto n = tree[curr[j]];
      mask &= ~(n.is_leaf() << j);
      if ((mask & (1 << j)) != 0) {
        curr[j] = tree.child_index<CATS_SUPPORTED>(n, curr[j], input[j * cols + n.fid()]);
      }
    }
  } while (mask != 0);

  // get the output from the leaves
  if (n_rows == NITEMS) {
    return tree_leaf_output<output_type, true>(tree, n_rows, curr);
  } else {
    return tree_leaf_output<output_type, false>(tree, n_rows, curr);
  }
}

template <typename output_type, typename tree_type>
__device__ __forceinline__ vec<1, output_type> infer_one_tree(
  tree_type tree, const typename tree_type::real_type* input, int cols, int rows)
{
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    bool cond = tree.child_index<true>(n, curr, input[n.fid()]);
    curr      = n.left(curr) + cond;
  }
  vec<1, output_type> out;
  /** dependent names are not considered templates by default,
      unless it's a member of a current [template] instantiation.**/
  out[0] = tree[curr].template output<output_type>();
  return out;
}

/**
The shared memory requirements for finalization stage may differ based
on the set of PTX architectures the kernels were compiled for, as well as
the CUDA compute capability of the device chosen for computation.

TODO (levsnv): run a test kernel during forest init to determine the compute capability
chosen for the inference, for an accurate sizeof(BlockReduce::TempStorage),
which is used in determining max NITEMS or max input data columns.

600 is the __CUDA_ARCH__ for Pascal (6.0) GPUs, which is not defined in
host code.
6.0 is the earliest compute capability supported by FIL and RAPIDS in general.
See https://rapids.ai/start.html as well as cmake defaults.
*/
// values below are defaults as of this change.
template <int NITEMS, typename real_t>
size_t block_reduce_footprint_host()
{
  return sizeof(
    typename cub::
      BlockReduce<vec<NITEMS, real_t>, FIL_TPB, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1, 600>::
        TempStorage);
}

template <int NITEMS, typename real_t>
size_t block_reduce_best_class_footprint_host()
{
  return sizeof(typename hipcub::BlockReduce<vec<NITEMS, best_margin_label<real_t>>,
                                          FIL_TPB,
                                          hipcub::BLOCK_REDUCE_WARP_REDUCTIONS,
                                          1,
                                          1,
                                          600>::TempStorage);
}

// the device template should achieve the best performance, using up-to-date
// CUB defaults
template <typename T, typename BinaryOp>
__device__ __forceinline__ T block_reduce(T value, BinaryOp op, void* storage)
{
  typedef hipcub::BlockReduce<T, FIL_TPB> BlockReduceT;
  return BlockReduceT(*(typename BlockReduceT::TempStorage*)storage).Reduce(value, op, blockDim.x);
}

template <int NITEMS,
          typename real_t,
          leaf_algo_t leaf_algo>  // = FLOAT_UNARY_BINARY
struct tree_aggregator_t {
  vec<NITEMS, real_t> acc;
  void* tmp_storage;

  /** shared memory footprint of the accumulator during
  the finalization of forest inference kernel, when infer_k output
  value is computed.
  num_classes is used for other template parameters */
  static size_t smem_finalize_footprint(size_t data_row_size,
                                        int num_classes,
                                        int log2_threads_per_tree,
                                        bool predict_proba)
  {
    return log2_threads_per_tree != 0 ? FIL_TPB * NITEMS * sizeof(real_t)
                                      : block_reduce_footprint_host<NITEMS, real_t>();
  }

  /** shared memory footprint of the accumulator during
  the accumulation of forest inference, when individual trees
  are inferred and partial aggregates are accumulated.
  num_classes is used for other template parameters */
  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  /**
  num_classes is used for other template parameters */
  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace,
                                               real_t* vector_leaf)
    : tmp_storage(finalize_workspace)
  {
  }

  __device__ __forceinline__ void accumulate(vec<NITEMS, real_t> single_tree_prediction,
                                             int tree,
                                             int thread_num_rows)
  {
    acc += single_tree_prediction;
  }

  __device__ INLINE_CONFIG void finalize(real_t* block_out,
                                         int block_num_rows,
                                         int output_stride,
                                         output_t transform,
                                         int num_trees,
                                         int log2_threads_per_tree)
  {
    if (FIL_TPB != 1 << log2_threads_per_tree) {  // anything to reduce?
      // ensure input columns can be overwritten (no threads traversing trees)
      __syncthreads();
      if (log2_threads_per_tree == 0) {
        acc = block_reduce(acc, vectorized(hipcub::Sum()), tmp_storage);
      } else {
        auto per_thread         = (vec<NITEMS, real_t>*)tmp_storage;
        per_thread[threadIdx.x] = acc;
        __syncthreads();
        // We have two pertinent cases for splitting FIL_TPB == 256 values:
        // 1. 2000 columns, which fit few threads/tree in shared memory,
        // so ~256 groups. These are the models that will run the slowest.
        // multi_sum performance is not sensitive to the radix here.
        // 2. 50 columns, so ~32 threads/tree, so ~8 groups. These are the most
        // popular.
        acc =
          multi_sum<5>(per_thread, 1 << log2_threads_per_tree, FIL_TPB >> log2_threads_per_tree);
      }
    }

    if (threadIdx.x * NITEMS >= block_num_rows) return;
    CUDA_PRAGMA_UNROLL
    for (int row = 0; row < NITEMS; ++row) {
      int out_preds_i = threadIdx.x * NITEMS + row;
      if (out_preds_i < block_num_rows) block_out[out_preds_i * output_stride] = acc[row];
    }
  }
};

// tmp_storage may overlap shared memory addressed by [begin, end)
// allreduce_shmem ensures no race conditions
template <typename Iterator, typename BinaryOp>
__device__ __forceinline__ auto allreduce_shmem(Iterator begin,
                                                Iterator end,
                                                BinaryOp op,
                                                void* tmp_storage)
{
  typedef typename std::iterator_traits<Iterator>::value_type value_type;
  value_type thread_partial;
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    thread_partial = op(thread_partial, *it);
  __syncthreads();  // free shared memory [begin, end)
  auto res = block_reduce(thread_partial, op, tmp_storage);
  // broadcast sum to all threads
  __syncthreads();  // free up tmp_storage
  if (threadIdx.x == 0) *(value_type*)tmp_storage = res;
  __syncthreads();
  return *(value_type*)tmp_storage;
}

// *begin and *end shall be struct vec
// tmp_storage may overlap shared memory addressed by [begin, end)
template <typename Iterator, typename real_t>
__device__ __forceinline__ void write_best_class(
  Iterator begin, Iterator end, void* tmp_storage, real_t* out, int num_rows)
{
  // reduce per-class candidate margins to one best class candidate
  // per thread (for each of the NITEMS rows)
  auto best = vec<begin->NITEMS, best_margin_label<real_t>>();
  for (int c = threadIdx.x; c < end - begin; c += blockDim.x)
    best = vectorized(hipcub::ArgMax())(best, to_vec(c, begin[c]));
  // [begin, end) may overlap tmp_storage
  __syncthreads();
  // find best class per block (for each of the NITEMS rows)
  best = block_reduce(best, vectorized(hipcub::ArgMax()), tmp_storage);
  // write it out to global memory
  if (threadIdx.x > 0) return;
  CUDA_PRAGMA_UNROLL
  for (int row = 0; row < best.NITEMS; ++row)
    if (row < num_rows) out[row] = best[row].key;
}

/// needed for softmax
struct shifted_exp {
  template <typename real_t>
  __device__ double operator()(real_t margin, real_t max) const
  {
    return exp(margin - max);
  }
};

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
template <typename Iterator>
__device__ __forceinline__ void block_softmax(Iterator begin, Iterator end, void* tmp_storage)
{
  // subtract max before exponentiating for numerical stability
  using value_type = typename std::iterator_traits<Iterator>::value_type;
  value_type max   = allreduce_shmem(begin, end, vectorized(hipcub::Max()), tmp_storage);
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    *it = vectorized(shifted_exp())(*it, max);
  // sum of exponents
  value_type soe = allreduce_shmem(begin, end, vectorized(hipcub::Sum()), tmp_storage);
  // softmax phase 2: normalization
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    *it /= soe;
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
template <typename Iterator, typename real_t>
__device__ __forceinline__ void normalize_softmax_and_write(Iterator begin,
                                                            Iterator end,
                                                            output_t transform,
                                                            int trees_per_class,
                                                            void* tmp_storage,
                                                            real_t* out,
                                                            int num_rows)
{
  if ((transform & output_t::AVG) != 0) {
    for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
      *it /= trees_per_class;
  }
  if ((transform & output_t::SOFTMAX) != 0) block_softmax(begin, end, tmp_storage);
  // write result to global memory
  CUDA_PRAGMA_UNROLL
  for (int row = 0; row < begin->NITEMS; ++row) {
    for (int c = threadIdx.x; c < end - begin; c += blockDim.x)
      if (row < num_rows) out[row * (end - begin) + c] = begin[c][row];
  }
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
// in case num_outputs > 1
template <typename Iterator, typename real_t>
__device__ __forceinline__ void class_margins_to_global_memory(Iterator begin,
                                                               Iterator end,
                                                               output_t transform,
                                                               int trees_per_class,
                                                               void* tmp_storage,
                                                               real_t* out,
                                                               int num_rows,
                                                               int num_outputs)
{
  if (num_outputs == 1) {  // will output class
    // reduce per-class candidate margins to one best class candidate
    // per thread (for each of the NITEMS rows)
    write_best_class(begin, end, tmp_storage, out, num_rows);
  } else {  // output softmax-ed margin
    normalize_softmax_and_write(begin, end, transform, trees_per_class, tmp_storage, out, num_rows);
  }
}

template <int NITEMS, typename real_t>
struct tree_aggregator_t<NITEMS, real_t, GROVE_PER_CLASS_FEW_CLASSES> {
  vec<NITEMS, real_t> acc;
  int num_classes;
  vec<NITEMS, real_t>* per_thread;
  void* tmp_storage;

  static size_t smem_finalize_footprint(size_t data_row_size,
                                        int num_classes,
                                        int log2_threads_per_tree,
                                        bool predict_proba)
  {
    size_t phase1 = (FIL_TPB - FIL_TPB % num_classes) * sizeof(vec<NITEMS, real_t>);
    size_t phase2 = predict_proba ? block_reduce_footprint_host<NITEMS, real_t>()
                                  : block_reduce_best_class_footprint_host<NITEMS, real_t>();
    return predict_proba ? phase1 + phase2 : std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace,
                                               real_t* vector_leaf)
    : num_classes(params.num_classes),
      per_thread((vec<NITEMS, real_t>*)finalize_workspace),
      tmp_storage(params.predict_proba ? per_thread + num_classes : finalize_workspace)
  {
  }

  __device__ __forceinline__ void accumulate(vec<NITEMS, real_t> single_tree_prediction,
                                             int tree,
                                             int thread_num_rows)
  {
    acc += single_tree_prediction;
  }

  __device__ INLINE_CONFIG void finalize(real_t* out,
                                         int num_rows,
                                         int num_outputs,
                                         output_t transform,
                                         int num_trees,
                                         int log2_threads_per_tree)
  {
    __syncthreads();  // free up input row in case it was in shared memory
    // load margin into shared memory
    per_thread[threadIdx.x] = acc;
    __syncthreads();
    acc = multi_sum<6>(per_thread, num_classes, blockDim.x / num_classes);
    if (threadIdx.x < num_classes) per_thread[threadIdx.x] = acc;
    __syncthreads();  // per_thread needs to be fully populated

    class_margins_to_global_memory(per_thread,
                                   per_thread + num_classes,
                                   transform,
                                   num_trees / num_classes,
                                   tmp_storage,
                                   out,
                                   num_rows,
                                   num_outputs);
  }
};

template <int NITEMS, typename real_t>
struct tree_aggregator_t<NITEMS, real_t, GROVE_PER_CLASS_MANY_CLASSES> {
  vec<NITEMS, real_t> acc;
  /// at first, per class margin, then, possibly, different softmax partials
  vec<NITEMS, real_t>* per_class_margin;
  void* tmp_storage;
  int num_classes;

  static size_t smem_finalize_footprint(size_t data_row_size,
                                        int num_classes,
                                        int log2_threads_per_tree,
                                        bool predict_proba)
  {
    size_t phase1 = data_row_size + smem_accumulate_footprint(num_classes);
    size_t phase2 = predict_proba ? block_reduce_footprint_host<NITEMS, real_t>()
                                  : block_reduce_best_class_footprint_host<NITEMS, real_t>();
    return predict_proba ? phase1 + phase2 : std::max(phase1, phase2);
  }

  static __host__ __device__ size_t smem_accumulate_footprint(int num_classes)
  {
    return num_classes * sizeof(vec<NITEMS, real_t>);
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace,
                                               real_t* vector_leaf)
    : per_class_margin((vec<NITEMS, real_t>*)accumulate_workspace),
      tmp_storage(params.predict_proba ? per_class_margin + num_classes : finalize_workspace),
      num_classes(params.num_classes)
  {
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      per_class_margin[c] = vec<NITEMS, real_t>(0);
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void accumulate(vec<NITEMS, real_t> single_tree_prediction,
                                             int tree,
                                             int thread_num_rows)
  {
    // since threads are assigned to consecutive classes, no need for atomics
    if (thread_num_rows > 0) { per_class_margin[tree % num_classes] += single_tree_prediction; }
    __syncthreads();
  }

  __device__ INLINE_CONFIG void finalize(real_t* out,
                                         int num_rows,
                                         int num_outputs,
                                         output_t transform,
                                         int num_trees,
                                         int log2_threads_per_tree)
  {
    class_margins_to_global_memory(per_class_margin,
                                   per_class_margin + num_classes,
                                   transform,
                                   num_trees / num_classes,
                                   tmp_storage,
                                   out,
                                   num_rows,
                                   num_outputs);
  }
};

template <int NITEMS, typename real_t>
struct tree_aggregator_t<NITEMS, real_t, VECTOR_LEAF> {
  // per_class_margin is a row-major matrix
  // of size num_threads_per_class * num_classes
  // used to acccumulate class values
  vec<NITEMS, real_t>* per_class_margin;
  vec<NITEMS, int>* vector_leaf_indices;
  int* thread_num_rows;
  int num_classes;
  int num_threads_per_class;
  real_t* vector_leaf;
  void* tmp_storage;

  static size_t smem_finalize_footprint(size_t data_row_size,
                                        int num_classes,
                                        int log2_threads_per_tree,
                                        bool predict_proba)
  {
    size_t phase1 = data_row_size + smem_accumulate_footprint(num_classes);
    size_t phase2 = predict_proba ? block_reduce_footprint_host<NITEMS, real_t>()
                                  : block_reduce_best_class_footprint_host<NITEMS, real_t>();
    return predict_proba ? phase1 + phase2 : std::max(phase1, phase2);
  }
  static size_t smem_accumulate_footprint(int num_classes)
  {
    return sizeof(vec<NITEMS, real_t>) * num_classes * max(1, FIL_TPB / num_classes) +
           sizeof(vec<NITEMS, int>) * FIL_TPB + sizeof(int) * FIL_TPB;
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace,
                                               real_t* vector_leaf)
    : num_classes(params.num_classes),
      num_threads_per_class(max(1, blockDim.x / params.num_classes)),
      vector_leaf(vector_leaf),
      tmp_storage(finalize_workspace)
  {
    // Assign workspace
    char* ptr        = (char*)accumulate_workspace;
    per_class_margin = (vec<NITEMS, real_t>*)ptr;
    ptr += sizeof(vec<NITEMS, real_t>) * num_classes * num_threads_per_class;
    vector_leaf_indices = (vec<NITEMS, int>*)ptr;
    ptr += sizeof(vec<NITEMS, int>) * blockDim.x;
    thread_num_rows = (int*)ptr;

    // Initialise shared memory
    for (int i = threadIdx.x; i < num_classes * num_threads_per_class; i += blockDim.x) {
      per_class_margin[i] = vec<NITEMS, real_t>();
    }
    vector_leaf_indices[threadIdx.x] = vec<NITEMS, int>();
    thread_num_rows[threadIdx.x]     = 0;
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void accumulate(vec<NITEMS, int> single_tree_prediction,
                                             int tree,
                                             int num_rows)
  {
    // Perform a transpose in shared memory
    // Assign each thread to a class, so they can accumulate without atomics
    __syncthreads();
    // Write indices to shared memory
    vector_leaf_indices[threadIdx.x] = single_tree_prediction;
    thread_num_rows[threadIdx.x]     = num_rows;
    __syncthreads();
    // i here refers to each element of the matrix per_class_margin
    for (int i = threadIdx.x; i < num_classes * num_threads_per_class; i += blockDim.x) {
      // if num_threads_per_class == 1, then c == i
      int c = i % num_classes;
      // iterate over original thread inputs with stride num_threads_per_class
      // j is the original thread input
      // we have num_classes threads for each j
      for (int j = i / num_classes; j < blockDim.x; j += num_threads_per_class) {
        for (int item = 0; item < thread_num_rows[j]; ++item) {
          real_t pred = vector_leaf[vector_leaf_indices[j][item] * num_classes + c];
          per_class_margin[i][item] += pred;
        }
      }
    }
  }
  __device__ INLINE_CONFIG void finalize(real_t* out,
                                         int num_rows,
                                         int num_outputs,
                                         output_t transform,
                                         int num_trees,
                                         int log2_threads_per_tree)
  {
    if (num_classes < blockDim.x) {
      __syncthreads();
      // Efficient implementation for small number of classes
      auto acc = multi_sum<6>(per_class_margin, num_classes, max(1, blockDim.x / num_classes));
      if (threadIdx.x < num_classes) per_class_margin[threadIdx.x] = acc;
      __syncthreads();
    }
    class_margins_to_global_memory(per_class_margin,
                                   per_class_margin + num_classes,
                                   transform,
                                   num_trees,
                                   tmp_storage,
                                   out,
                                   num_rows,
                                   num_outputs);
  }
};

template <int NITEMS, typename real_t>
struct tree_aggregator_t<NITEMS, real_t, CATEGORICAL_LEAF> {
  // could switch to uint16_t to save shared memory
  // provided raft::myAtomicAdd(short*) simulated with appropriate shifts
  int* votes;
  int num_classes;

  static size_t smem_finalize_footprint(size_t data_row_size,
                                        int num_classes,
                                        int log2_threads_per_tree,
                                        bool predict_proba)
  {
    // not accounting for lingering accumulate_footprint during finalize()
    return 0;
  }
  static size_t smem_accumulate_footprint(int num_classes)
  {
    return sizeof(int) * num_classes * NITEMS;
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace,
                                               real_t* vector_leaf)
    : num_classes(params.num_classes), votes((int*)accumulate_workspace)
  {
    for (int c = threadIdx.x; c < num_classes; c += FIL_TPB * NITEMS)
      CUDA_PRAGMA_UNROLL
    for (int item = 0; item < NITEMS; ++item)
      votes[c * NITEMS + item] = 0;
    // __syncthreads() is called in infer_k
  }
  __device__ __forceinline__ void accumulate(vec<NITEMS, int> single_tree_prediction,
                                             int tree,
                                             int thread_num_rows)
  {
    if (thread_num_rows == 0) return;
    CUDA_PRAGMA_UNROLL
    for (int item = 0; item < NITEMS; ++item) {
      raft::myAtomicAdd(votes + single_tree_prediction[item] * NITEMS + item, 1);
    }
  }
  // class probabilities or regression. for regression, num_classes
  // is just the number of outputs for each data instance
  __device__ __forceinline__ void finalize_multiple_outputs(real_t* out, int num_rows)
  {
    __syncthreads();
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x) {
      CUDA_PRAGMA_UNROLL
      for (int row = 0; row < num_rows; ++row)
        out[row * num_classes + c] = votes[c * NITEMS + row];
    }
  }
  // using this when predicting a single class label, as opposed to sparse class vector
  // or class probabilities or regression
  __device__ __forceinline__ void finalize_class_label(real_t* out, int num_rows)
  {
    __syncthreads();  // make sure all votes[] are final
    int item = threadIdx.x;
    int row  = item;
    if (item < NITEMS && row < num_rows) {
      int max_votes  = 0;
      int best_class = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (votes[c * NITEMS + item] > max_votes) {
          max_votes  = votes[c * NITEMS + item];
          best_class = c;
        }
      }
      out[row] = best_class;
    }
  }
  __device__ INLINE_CONFIG void finalize(real_t* out,
                                         int num_rows,
                                         int num_outputs,
                                         output_t transform,
                                         int num_trees,
                                         int log2_threads_per_tree)
  {
    if (num_outputs > 1) {
      // only supporting num_outputs == num_classes
      finalize_multiple_outputs(out, num_rows);
    } else {
      finalize_class_label(out, num_rows);
    }
  }
};

template <typename real_t>
__device__ INLINE_CONFIG void load_data(real_t* sdata,
                                        const real_t* block_input,
                                        predict_params params,
                                        int rows_per_block,
                                        int block_num_rows)
{
  int num_cols     = params.num_cols;
  int sdata_stride = params.sdata_stride();
  // cache the row for all threads to reuse
  // 2021: latest SMs still do not have >256KiB of shared memory/block required to
  // exceed the uint16_t
  CUDA_PRAGMA_UNROLL
  for (uint16_t input_idx = threadIdx.x; input_idx < block_num_rows * num_cols;
       input_idx += blockDim.x) {
    // for even num_cols, we need to pad sdata_stride to reduce bank conflicts
    // assuming here that sdata_stride == num_cols + 1
    // then, idx / num_cols * sdata_stride + idx % num_cols == idx + idx / num_cols
    uint16_t sdata_idx =
      sdata_stride == num_cols ? input_idx : input_idx + input_idx / (uint16_t)num_cols;
    sdata[sdata_idx] = block_input[input_idx];
  }
  CUDA_PRAGMA_UNROLL
  for (int idx = block_num_rows * sdata_stride; idx < rows_per_block * sdata_stride;
       idx += blockDim.x)
    sdata[idx] = 0.0f;
}

template <int NITEMS,
          leaf_algo_t leaf_algo,
          bool cols_in_shmem,
          bool CATS_SUPPORTED,
          class storage_type>
__global__ void infer_k(storage_type forest, predict_params params)
{
  using real_t = typename storage_type::real_type;
  extern __shared__ char smem[];
  real_t* sdata      = reinterpret_cast<real_t*>(smem);
  int sdata_stride   = params.sdata_stride();
  int rows_per_block = NITEMS << params.log2_threads_per_tree;
  int num_cols       = params.num_cols;
  int thread_row0    = NITEMS * modpow2(threadIdx.x, params.log2_threads_per_tree);
  for (int64_t block_row0 = blockIdx.x * rows_per_block; block_row0 < params.num_rows;
       block_row0 += rows_per_block * gridDim.x) {
    int block_num_rows =
      max(0, (int)min((int64_t)rows_per_block, (int64_t)params.num_rows - block_row0));
    const real_t* block_input =
      reinterpret_cast<const real_t*>(params.data) + block_row0 * num_cols;
    if constexpr (cols_in_shmem)
      load_data(sdata, block_input, params, rows_per_block, block_num_rows);

    tree_aggregator_t<NITEMS, real_t, leaf_algo> acc(
      params, (char*)sdata + params.cols_shmem_size(), sdata, forest.vector_leaf_);

    __syncthreads();  // for both row cache init and acc init
    // one block works on NITEMS * threads_per_tree rows and the whole forest
    // one thread works on NITEMS rows

    int thread_tree0    = threadIdx.x >> params.log2_threads_per_tree;
    int tree_stride     = blockDim.x >> params.log2_threads_per_tree;
    int thread_num_rows = max(0, min(NITEMS, block_num_rows - thread_row0));
    for (int tree = thread_tree0; tree - thread_tree0 < forest.num_trees(); tree += tree_stride) {
      /* tree - thread_tree0 < forest.num_trees() is a necessary but block-uniform
         condition for "tree < forest.num_trees()". It lets use __syncthreads()
         and is made exact below.
         Same with thread_num_rows > 0
      */
      using pred_t = typename leaf_output_t<leaf_algo, real_t>::T;
      vec<NITEMS, pred_t> prediction;
      if (tree < forest.num_trees() && thread_num_rows != 0) {
        prediction = infer_one_tree<NITEMS, CATS_SUPPORTED, pred_t>(
          forest[tree],
          cols_in_shmem ? sdata + thread_row0 * sdata_stride : block_input + thread_row0 * num_cols,
          cols_in_shmem ? sdata_stride : num_cols,
          cols_in_shmem ? NITEMS : thread_num_rows);
      }
      // All threads must enter accumulate
      // Dummy threads can be marked as having 0 rows
      acc.accumulate(prediction, tree, tree < forest.num_trees() ? thread_num_rows : 0);
    }
    acc.finalize(reinterpret_cast<real_t*>(params.preds) + params.num_outputs * block_row0,
                 block_num_rows,
                 params.num_outputs,
                 params.transform,
                 forest.num_trees(),
                 params.log2_threads_per_tree);
    __syncthreads();  // free up acc's shared memory resources for next row set
  }
}

template <int NITEMS, typename real_t, leaf_algo_t leaf_algo>
size_t shmem_size_params::get_smem_footprint()
{
  size_t finalize_footprint = tree_aggregator_t<NITEMS, real_t, leaf_algo>::smem_finalize_footprint(
    cols_shmem_size(), num_classes, log2_threads_per_tree, predict_proba);
  size_t accumulate_footprint =
    tree_aggregator_t<NITEMS, real_t, leaf_algo>::smem_accumulate_footprint(num_classes) +
    cols_shmem_size();
  return std::max(accumulate_footprint, finalize_footprint);
}

template <class KernelParams>
int compute_smem_footprint::run(predict_params ssp)
{
  switch (ssp.sizeof_real) {
    case 4:
      return ssp
        .template get_smem_footprint<KernelParams::N_ITEMS, float, KernelParams::LEAF_ALGO>();
    case 8:
      return ssp
        .template get_smem_footprint<KernelParams::N_ITEMS, double, KernelParams::LEAF_ALGO>();
    default:
      ASSERT(false,
             "internal error: sizeof_real == %d, but must be 4 or 8",
             static_cast<int>(ssp.sizeof_real));
      // unreachable
      return 0;
  }
}

// make sure to instantiate all possible get_smem_footprint instantiations
template int dispatch_on_fil_template_params(compute_smem_footprint, predict_params);

template <typename storage_type>
struct infer_k_storage_template : dispatch_functor<void> {
  storage_type forest;
  hipStream_t stream;
  infer_k_storage_template(storage_type forest_, hipStream_t stream_)
    : forest(forest_), stream(stream_)
  {
  }

  template <class KernelParams = KernelTemplateParams<>>
  void run(predict_params params)
  {
    params.num_blocks = params.num_blocks != 0
                          ? params.num_blocks
                          : raft::ceildiv(int(params.num_rows), params.n_items);
    infer_k<KernelParams::N_ITEMS,
            KernelParams::LEAF_ALGO,
            KernelParams::COLS_IN_SHMEM,
            KernelParams::CATS_SUPPORTED>
      <<<params.num_blocks, params.block_dim_x, params.shm_sz, stream>>>(forest, params);
    RAFT_CUDA_TRY(hipPeekAtLastError());
  }
};

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream)
{
  dispatch_on_fil_template_params(infer_k_storage_template<storage_type>(forest, stream), params);
}

template void infer<dense_storage_f32>(dense_storage_f32 forest,
                                       predict_params params,
                                       hipStream_t stream);
template void infer<dense_storage_f64>(dense_storage_f64 forest,
                                       predict_params params,
                                       hipStream_t stream);
template void infer<sparse_storage16_f32>(sparse_storage16_f32 forest,
                                          predict_params params,
                                          hipStream_t stream);
template void infer<sparse_storage16_f64>(sparse_storage16_f64 forest,
                                          predict_params params,
                                          hipStream_t stream);
template void infer<sparse_storage8>(sparse_storage8 forest,
                                     predict_params params,
                                     hipStream_t stream);

}  // namespace fil
}  // namespace ML
