#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>

#include <thrust/functional.h>
#include <cuml/fil/multi_sum.cuh>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;

// vec wraps float[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec;

template <typename BinaryOp>
struct Vectorized {
  BinaryOp op;
  __device__ Vectorized(BinaryOp op_) : op(op_) {}
  template <int NITEMS, typename T>
  constexpr __host__ __device__ __forceinline__ vec<NITEMS, T> operator()(
    vec<NITEMS, T> a, vec<NITEMS, T> b) const {
    vec<NITEMS, T> c;
#pragma unroll
    for (int i = 0; i < NITEMS; i++) c[i] = op(a[i], b[i]);
    return c;
  }
};
template <typename BinaryOp>
constexpr __host__ __device__ Vectorized<BinaryOp> vectorized(BinaryOp op) {
  return op;
}

template <int N, typename T>
struct vec {
  static const int NITEMS = N;
  T data[N];
  explicit __host__ __device__ vec(T t) {
#pragma unroll
    for (int i = 0; i < N; ++i) data[i] = t;
  }
  __host__ __device__ vec() : vec(T()) {}
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a,
                                                 const vec<N, T>& b) {
    return vectorized(hipcub::Sum())(a, b);
  }
  friend __host__ __device__ void operator+=(vec<N, T>& a, const vec<N, T>& b) {
    a = a + b;
  }
  template <typename Vec>
  friend __host__ __device__ vec<N, T> operator/(vec<N, T>& a, const Vec& b) {
    return vectorized(thrust::divides<T>())(a, vec<N, T>(b));
  }
  template <typename Vec>
  friend __host__ __device__ void operator/=(vec<N, T>& a, const Vec& b) {
    a = a / b;
  }
};

struct best_margin_label : hipcub::KeyValuePair<int, float> {
  __host__ __device__ best_margin_label(hipcub::KeyValuePair<int, float> pair)
    : hipcub::KeyValuePair<int, float>(pair) {}
  __host__ __device__ best_margin_label(int c = 0, float f = -INFINITY)
    : hipcub::KeyValuePair<int, float>({c, f}) {}
};

template <int NITEMS>
__device__ __forceinline__ vec<NITEMS, best_margin_label> to_vec(
  int c, vec<NITEMS, float> margin) {
  vec<NITEMS, best_margin_label> ret;
#pragma unroll
  for (int i = 0; i < NITEMS; ++i) ret[i] = best_margin_label(c, margin[i]);
  return ret;
}

struct ArgMax {
  template <int NITEMS>
  __host__ __device__ __forceinline__ vec<NITEMS, best_margin_label> operator()(
    vec<NITEMS, best_margin_label> a, vec<NITEMS, best_margin_label> b) const {
    vec<NITEMS, best_margin_label> c;
#pragma unroll
    for (int i = 0; i < NITEMS; i++) c[i] = hipcub::ArgMax()(a[i], b[i]);
    return c;
  }
};

/** tree_leaf_output returns the leaf outputs from the tree with leaf indices
    given by leaves for n_rows items. FULL_ITEMS indicates whether n_rows ==
    NITEMS, to allow the compiler to skip the conditional when unrolling the
    loop. */
template <typename output_type, bool FULL_NITEMS, int NITEMS,
          typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> tree_leaf_output(
  tree_type tree, int n_rows, int (&leaves)[NITEMS]) {
  vec<NITEMS, output_type> out(0);
#pragma unroll
  for (int j = 0; j < NITEMS; ++j) {
    if (FULL_NITEMS || j < n_rows) {
      /** dependent names are not considered templates by default, unless it's a
          member of a current [template] instantiation. As output<>() is a
          member function inherited from the base class, template
          output<output_type>() is required. */
      out[j] = tree[leaves[j]].template output<output_type>();
    }
  }
  return out;
}

template <int NITEMS, typename output_type, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> infer_one_tree(
  tree_type tree, const float* input, int cols, int n_rows) {
  // find the leaf nodes for each row
  int curr[NITEMS];
  // the first n_rows are active
  int mask = (1 << n_rows) - 1;
  for (int j = 0; j < NITEMS; ++j) curr[j] = 0;
  do {
#pragma unroll
    for (int j = 0; j < NITEMS; ++j) {
      auto n = tree[curr[j]];
      mask &= ~(n.is_leaf() << j);
      if ((mask & (1 << j)) != 0) {
        float val = input[j * cols + n.fid()];
        bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
        curr[j] = n.left(curr[j]) + cond;
      }
    }
  } while (mask != 0);

  // get the output from the leaves
  if (n_rows == NITEMS) {
    return tree_leaf_output<output_type, true>(tree, n_rows, curr);
  } else {
    return tree_leaf_output<output_type, false>(tree, n_rows, curr);
  }
}

template <typename output_type, typename tree_type>
__device__ __forceinline__ vec<1, output_type> infer_one_tree(
  tree_type tree, const float* input, int cols, int rows) {
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    float val = input[n.fid()];
    bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
    curr = n.left(curr) + cond;
  }
  vec<1, output_type> out;
  out[0] = tree[curr].base_node::output<output_type>();
  return out;
}

/**
The shared memory requirements for finalization stage may differ based
on the set of PTX architectures the kernels were compiled for, as well as 
the CUDA compute capability of the device chosen for computation.

TODO (levsnv): run a test kernel during forest init to determine the compute capability
chosen for the inference, for an accurate sizeof(BlockReduce::TempStorage),
which is used in determining max NITEMS or max input data columns.

600 is the __CUDA_ARCH__ for Pascal (6.0) GPUs, which is not defined in
host code.
6.0 is the earliest compute capability supported by FIL and RAPIDS in general.
See https://rapids.ai/start.html as well as cmake defaults.
*/
// values below are defaults as of this change.
template <int NITEMS>
size_t block_reduce_footprint_host() {
  return sizeof(typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB,
                                          hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1,
                                          1, 600>::TempStorage);
}

template <int NITEMS>
size_t block_reduce_best_class_footprint_host() {
  return sizeof(
    typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB,
                              hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1,
                              600>::TempStorage);
}

// the device template should achieve the best performance, using up-to-date
// CUB defaults
template <typename T, typename BinaryOp>
__device__ __forceinline__ T block_reduce(T value, BinaryOp op, void* storage) {
  typedef hipcub::BlockReduce<T, FIL_TPB> BlockReduceT;
  return BlockReduceT(*(typename BlockReduceT::TempStorage*)storage)
    .Reduce(value, op, blockDim.x);
}

template <int NITEMS,
          leaf_algo_t leaf_algo>  // = FLOAT_UNARY_BINARY
struct tree_aggregator_t {
  vec<NITEMS, float> acc;
  void* tmp_storage;

  /** shared memory footprint of the accumulator during
  the finalization of forest inference kernel, when infer_k output
  value is computed.
  num_classes is used for other template parameters */
  static size_t smem_finalize_footprint(size_t data_row_size, int num_classes,
                                        bool predict_proba) {
    return block_reduce_footprint_host<NITEMS>();
  }

  /** shared memory footprint of the accumulator during
  the accumulation of forest inference, when individual trees
  are inferred and partial aggregates are accumulated.
  num_classes is used for other template parameters */
  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  /** 
  num_classes is used for other template parameters */
  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : tmp_storage(finalize_workspace) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree, int num_rows) {
    acc += single_tree_prediction;
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int output_stride,
                                           output_t transform, int num_trees) {
    __syncthreads();
    acc = block_reduce(acc, vectorized(hipcub::Sum()), tmp_storage);
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < NITEMS; ++row)
      if (row < num_rows) out[row * output_stride] = acc[row];
  }
};

// tmp_storage may overlap shared memory addressed by [begin, end)
// allreduce_shmem ensures no race conditions
template <typename Iterator, typename BinaryOp>
__device__ __forceinline__ auto allreduce_shmem(Iterator begin, Iterator end,
                                                BinaryOp op,
                                                void* tmp_storage) {
  typedef typename std::iterator_traits<Iterator>::value_type value_type;
  value_type thread_partial;
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    thread_partial = op(thread_partial, *it);
  __syncthreads();  // free shared memory [begin, end)
  auto res = block_reduce(thread_partial, op, tmp_storage);
  // broadcast sum to all threads
  __syncthreads();  // free up tmp_storage
  if (threadIdx.x == 0) *(value_type*)tmp_storage = res;
  __syncthreads();
  return *(value_type*)tmp_storage;
}

// *begin and *end shall be struct vec
// tmp_storage may overlap shared memory addressed by [begin, end)
template <typename Iterator>
__device__ __forceinline__ void write_best_class(Iterator begin, Iterator end,
                                                 void* tmp_storage, float* out,
                                                 int num_rows) {
  // reduce per-class candidate margins to one best class candidate
  // per thread (for each of the NITEMS rows)
  auto best = vec<begin->NITEMS, best_margin_label>();
  for (int c = threadIdx.x; c < end - begin; c += blockDim.x)
    best = vectorized(hipcub::ArgMax())(best, to_vec(c, begin[c]));
  // [begin, end) may overlap tmp_storage
  __syncthreads();
  // find best class per block (for each of the NITEMS rows)
  best = block_reduce(best, vectorized(hipcub::ArgMax()), tmp_storage);
  // write it out to global memory
  if (threadIdx.x > 0) return;
#pragma unroll
  for (int row = 0; row < best.NITEMS; ++row)
    if (row < num_rows) out[row] = best[row].key;
}

/// needed for softmax
__device__ float shifted_exp(float margin, float max) {
  return expf(margin - max);
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
template <typename Iterator>
__device__ __forceinline__ void block_softmax(Iterator begin, Iterator end,
                                              void* tmp_storage) {
  // subtract max before exponentiating for numerical stability
  typedef typename std::iterator_traits<Iterator>::value_type value_type;
  value_type max =
    allreduce_shmem(begin, end, vectorized(hipcub::Max()), tmp_storage);
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    *it = vectorized(shifted_exp)(*it, max);
  // sum of exponents
  value_type soe =
    allreduce_shmem(begin, end, vectorized(hipcub::Sum()), tmp_storage);
  // softmax phase 2: normalization
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    *it /= soe;
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
template <typename Iterator>
__device__ __forceinline__ void normalize_softmax_and_write(
  Iterator begin, Iterator end, output_t transform, int trees_per_class,
  void* tmp_storage, float* out, int num_rows) {
  if ((transform & output_t::AVG) != 0) {
    for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
      *it /= trees_per_class;
  }
  if ((transform & output_t::SOFTMAX) != 0)
    block_softmax(begin, end, tmp_storage);
// write result to global memory
#pragma unroll
  for (int row = 0; row < begin->NITEMS; ++row) {
    for (int c = threadIdx.x; c < end - begin; c += blockDim.x)
      if (row < num_rows) out[row * (end - begin) + c] = begin[c][row];
  }
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
// in case num_outputs > 1
template <typename Iterator>
__device__ __forceinline__ void class_margins_to_global_memory(
  Iterator begin, Iterator end, output_t transform, int trees_per_class,
  void* tmp_storage, float* out, int num_rows, int num_outputs) {
  if (num_outputs == 1) {  // will output class
    // reduce per-class candidate margins to one best class candidate
    // per thread (for each of the NITEMS rows)
    write_best_class(begin, end, tmp_storage, out, num_rows);
  } else {  // output softmax-ed margin
    normalize_softmax_and_write(begin, end, transform, trees_per_class,
                                tmp_storage, out, num_rows);
  }
}

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_FEW_CLASSES> {
  vec<NITEMS, float> acc;
  int num_classes;
  vec<NITEMS, float>* per_thread;
  void* tmp_storage;

  static size_t smem_finalize_footprint(size_t data_row_size, int num_classes,
                                        bool predict_proba) {
    size_t phase1 =
      (FIL_TPB - FIL_TPB % num_classes) * sizeof(vec<NITEMS, float>);
    size_t phase2 = predict_proba
                      ? block_reduce_footprint_host<NITEMS>()
                      : block_reduce_best_class_footprint_host<NITEMS>();
    return predict_proba ? phase1 + phase2 : std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : num_classes(params.num_classes),
      per_thread((vec<NITEMS, float>*)finalize_workspace),
      tmp_storage(params.predict_proba ? per_thread + num_classes
                                       : finalize_workspace) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree, int num_rows) {
    acc += single_tree_prediction;
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs, output_t transform,
                                           int num_trees) {
    __syncthreads();  // free up input row in case it was in shared memory
    // load margin into shared memory
    per_thread[threadIdx.x] = acc;
    __syncthreads();
    acc = multi_sum<6>(per_thread, num_classes, blockDim.x / num_classes);
    if (threadIdx.x < num_classes) per_thread[threadIdx.x] = acc;
    __syncthreads();  // per_thread needs to be fully populated

    class_margins_to_global_memory(per_thread, per_thread + num_classes,
                                   transform, num_trees / num_classes,
                                   tmp_storage, out, num_rows, num_outputs);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_MANY_CLASSES> {
  vec<NITEMS, float> acc;
  /// at first, per class margin, then, possibly, different softmax partials
  vec<NITEMS, float>* per_class_margin;
  void* tmp_storage;
  int num_classes;

  static size_t smem_finalize_footprint(size_t data_row_size, int num_classes,
                                        bool predict_proba) {
    size_t phase1 = data_row_size + smem_accumulate_footprint(num_classes);
    size_t phase2 = predict_proba
                      ? block_reduce_footprint_host<NITEMS>()
                      : block_reduce_best_class_footprint_host<NITEMS>();
    return predict_proba ? phase1 + phase2 : std::max(phase1, phase2);
  }

  static __host__ __device__ size_t smem_accumulate_footprint(int num_classes) {
    return num_classes * sizeof(vec<NITEMS, float>);
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : per_class_margin((vec<NITEMS, float>*)accumulate_workspace),
      tmp_storage(params.predict_proba ? per_class_margin + num_classes
                                       : finalize_workspace),
      num_classes(params.num_classes) {
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      per_class_margin[c] = vec<NITEMS, float>(0);
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree, int num_rows) {
    // since threads are assigned to consecutive classes, no need for atomics
    per_class_margin[tree % num_classes] += single_tree_prediction;
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs, output_t transform,
                                           int num_trees) {
    class_margins_to_global_memory(
      per_class_margin, per_class_margin + num_classes, transform,
      num_trees / num_classes, tmp_storage, out, num_rows, num_outputs);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, CATEGORICAL_LEAF> {
  // could switch to unsigned short to save shared memory
  // provided raft::myAtomicAdd(short*) simulated with appropriate shifts
  int* votes;
  int num_classes;

  static size_t smem_finalize_footprint(size_t data_row_size, int num_classes,
                                        bool predict_proba) {
    // not accounting for lingering accumulate_footprint during finalize()
    return 0;
  }
  static size_t smem_accumulate_footprint(int num_classes) {
    return sizeof(int) * num_classes * NITEMS;
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : num_classes(params.num_classes), votes((int*)accumulate_workspace) {
    for (int c = threadIdx.x; c < num_classes; c += FIL_TPB * NITEMS)
#pragma unroll
      for (int item = 0; item < NITEMS; ++item) votes[c * NITEMS + item] = 0;
    // __syncthreads() is called in infer_k
  }
  __device__ __forceinline__ void accumulate(
    vec<NITEMS, int> single_tree_prediction, int tree, int num_rows) {
#pragma unroll
    for (int item = 0; item < NITEMS; ++item)
      raft::myAtomicAdd(votes + single_tree_prediction[item] * NITEMS + item,
                        1);
  }
  // class probabilities or regression. for regression, num_classes
  // is just the number of outputs for each data instance
  __device__ __forceinline__ void finalize_multiple_outputs(float* out,
                                                            int num_rows) {
    __syncthreads();
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x) {
#pragma unroll
      for (int row = 0; row < num_rows; ++row)
        out[row * num_classes + c] = votes[c * NITEMS + row];
    }
  }
  // using this when predicting a single class label, as opposed to sparse class vector
  // or class probabilities or regression
  __device__ __forceinline__ void finalize_class_label(float* out,
                                                       int num_rows) {
    __syncthreads();
    int item = threadIdx.x;
    int row = item;
    if (item < NITEMS && row < num_rows) {
      int max_votes = 0;
      int best_class = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (votes[c * NITEMS + item] > max_votes) {
          max_votes = votes[c * NITEMS + item];
          best_class = c;
        }
      }
      out[row] = best_class;
    }
  }
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs, output_t transform,
                                           int num_trees) {
    if (num_outputs > 1) {
      // only supporting num_outputs == num_classes
      finalize_multiple_outputs(out, num_rows);
    } else {
      finalize_class_label(out, num_rows);
    }
  }
};

template <int NITEMS, leaf_algo_t leaf_algo, bool cols_in_shmem,
          class storage_type>
__global__ void infer_k(storage_type forest, predict_params params) {
  extern __shared__ char smem[];
  float* sdata = (float*)smem;
  int num_cols = params.num_cols;
  for (size_t block_row0 = blockIdx.x * NITEMS; block_row0 < params.num_rows;
       block_row0 += NITEMS * gridDim.x) {
    size_t num_input_rows = min((size_t)NITEMS, params.num_rows - block_row0);
    const float* block_input = params.data + block_row0 * num_cols;
    if (cols_in_shmem) {
      // cache the row for all threads to reuse
      size_t feature = 0;
#pragma unroll
      for (feature = threadIdx.x; feature < num_input_rows * num_cols;
           feature += blockDim.x)
        sdata[feature] = block_input[feature];
#pragma unroll
      for (; feature < NITEMS * num_cols; feature += blockDim.x)
        sdata[feature] = 0.0f;
    }

    tree_aggregator_t<NITEMS, leaf_algo> acc(
      params, (char*)sdata + params.cols_shmem_size(), sdata);

    __syncthreads();  // for both row cache init and acc init

    // one block works on NITEMS rows and the whole forest
    for (int j = threadIdx.x; j - threadIdx.x < forest.num_trees();
         j += blockDim.x) {
      /* j - threadIdx.x < forest.num_trees() is a necessary but block-uniform
         condition for "j < forest.num_trees()". It lets use __syncthreads()
         and is made exact below.
      */
      if (j < forest.num_trees()) {
        acc.accumulate(infer_one_tree<NITEMS, leaf_output_t<leaf_algo>::T>(
                         forest[j], cols_in_shmem ? sdata : block_input,
                         num_cols, num_input_rows),
                       j, num_input_rows);
      }
      if (leaf_algo == GROVE_PER_CLASS_MANY_CLASSES) __syncthreads();
    }
    acc.finalize(params.preds + params.num_outputs * block_row0, num_input_rows,
                 params.num_outputs, params.transform, forest.num_trees());
    __syncthreads();  // free up acc's shared memory resources for next row set
  }
}

template <int NITEMS, leaf_algo_t leaf_algo>
size_t shmem_size_params::get_smem_footprint() {
  size_t finalize_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_finalize_footprint(
      cols_shmem_size(), num_classes, predict_proba);
  size_t accumulate_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_accumulate_footprint(
      num_classes) +
    cols_shmem_size();

  return std::max(accumulate_footprint, finalize_footprint);
}

template <int NITEMS>
size_t shmem_size_params::get_smem_footprint() {
  switch (leaf_algo) {
    case FLOAT_UNARY_BINARY:
      return get_smem_footprint<NITEMS, FLOAT_UNARY_BINARY>();
    case CATEGORICAL_LEAF:
      return get_smem_footprint<NITEMS, CATEGORICAL_LEAF>();
    case GROVE_PER_CLASS:
      if (num_classes > FIL_TPB)
        return get_smem_footprint<NITEMS, GROVE_PER_CLASS_MANY_CLASSES>();
      return get_smem_footprint<NITEMS, GROVE_PER_CLASS_FEW_CLASSES>();
    default:
      ASSERT(false, "internal error: unexpected leaf_algo_t");
  }
}

void shmem_size_params::compute_smem_footprint() {
  switch (n_items) {
    case 1:
      shm_sz = get_smem_footprint<1>();
      break;
    case 2:
      shm_sz = get_smem_footprint<2>();
      break;
    case 3:
      shm_sz = get_smem_footprint<3>();
      break;
    case 4:
      shm_sz = get_smem_footprint<4>();
      break;
    default:
      ASSERT(false, "internal error: n_items > 4");
  }
}

template <leaf_algo_t leaf_algo, bool cols_in_shmem, typename storage_type>
void infer_k_nitems_launcher(storage_type forest, predict_params params,
                             hipStream_t stream, int block_dim_x) {
  switch (params.n_items) {
    case 1:
      infer_k<1, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    case 2:
      infer_k<2, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    case 3:
      infer_k<3, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    case 4:
      infer_k<4, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    default:
      ASSERT(false, "internal error: nitems > 4");
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template <leaf_algo_t leaf_algo, typename storage_type>
void infer_k_launcher(storage_type forest, predict_params params,
                      hipStream_t stream, int blockdim_x) {
  params.num_blocks = params.num_blocks != 0
                        ? params.num_blocks
                        : raft::ceildiv(int(params.num_rows), params.n_items);
  if (params.cols_in_shmem) {
    infer_k_nitems_launcher<leaf_algo, true>(forest, params, stream,
                                             blockdim_x);
  } else {
    infer_k_nitems_launcher<leaf_algo, false>(forest, params, stream,
                                              blockdim_x);
  }
}

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream) {
  switch (params.leaf_algo) {
    case FLOAT_UNARY_BINARY:
      infer_k_launcher<FLOAT_UNARY_BINARY>(forest, params, stream, FIL_TPB);
      break;
    case GROVE_PER_CLASS:
      if (params.num_classes > FIL_TPB) {
        params.leaf_algo = GROVE_PER_CLASS_MANY_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_MANY_CLASSES>(forest, params, stream,
                                                       FIL_TPB);
      } else {
        params.leaf_algo = GROVE_PER_CLASS_FEW_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_FEW_CLASSES>(
          forest, params, stream, FIL_TPB - FIL_TPB % params.num_classes);
      }
      break;
    case CATEGORICAL_LEAF:
      infer_k_launcher<CATEGORICAL_LEAF>(forest, params, stream, FIL_TPB);
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_algo");
  }
}

template void infer<dense_storage>(dense_storage forest, predict_params params,
                                   hipStream_t stream);
template void infer<sparse_storage16>(sparse_storage16 forest,
                                      predict_params params,
                                      hipStream_t stream);
template void infer<sparse_storage8>(sparse_storage8 forest,
                                     predict_params params,
                                     hipStream_t stream);

}  // namespace fil
}  // namespace ML
