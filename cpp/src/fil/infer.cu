#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;

// vec wraps float[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec {
  T data[N];
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ void operator+=(vec<N, T>& a,
                                             const vec<N, T>& b) {
#pragma unroll
    for (int i = 0; i < N; ++i) a[i] += b[i];
  }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a,
                                                 const vec<N, T>& b) {
    vec<N, T> r = a;
    r += b;
    return r;
  }                                               
};

template <int NITEMS, typename TOUTPUT, typename tree_type>
__device__ __forceinline__ vec<NITEMS, TOUTPUT> infer_one_tree(tree_type tree, float* sdata,
                                               int cols) {
  int curr[NITEMS];
  int mask = (1 << NITEMS) - 1;  // all active
  for (int j = 0; j < NITEMS; ++j) curr[j] = 0;
  do {
#pragma unroll
    for (int j = 0; j < NITEMS; ++j) {
      if ((mask >> j) & 1 == 0) continue;
      auto n = tree[curr[j]];
      if (n.is_leaf()) {
        mask &= ~(1 << j);
        continue;
      }
      float val = sdata[j * cols + n.fid()];
      bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
      curr[j] = n.left(curr[j]) + cond;
    }
  } while (mask != 0);
  vec<NITEMS, TOUTPUT> out;
#pragma unroll
  for (int j = 0; j < NITEMS; ++j)
    out[j] = tree[curr[j]].base_node::output<TOUTPUT>();
  return out;
}

template <typename TOUTPUT, typename tree_type>
__device__ __forceinline__ vec<1, TOUTPUT> infer_one_tree(tree_type tree, float* sdata,
                                               int cols) {
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    float val = sdata[n.fid()];
    bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
    curr = n.left(curr) + cond;
  }
  vec<1, TOUTPUT> out;
  out[0] = tree[curr].base_node::output<TOUTPUT>();
  return out;
}

template <int NITEMS, leaf_value_t leaf_payload_type, typename TNODE_PAYLOAD>
class AggregateTrees {
  public:
    __device__ __forceinline__ AggregateTrees(int num_output_classes, void* smem_workspace);
    __device__ __forceinline__ void accumulate(vec<NITEMS, TNODE_PAYLOAD> out);
    __device__ __forceinline__ void finalize(float* out, int num_rows);
};

template <int NITEMS> class AggregateTrees<NITEMS, FLOAT_SCALAR, float> {
  vec<NITEMS, float> acc;
  int num_output_classes;
  public:
    __device__ __forceinline__ AggregateTrees(int num_output_classes_, void*):
    num_output_classes(num_output_classes_) {
      // TODO: even if num_output_classes == 2, in regression, this needs to change
      #pragma unroll
      for (int i = 0; i < NITEMS; ++i) acc[i] = 0.0f;
    }
    __device__ __forceinline__ void accumulate(vec<NITEMS, float> out) {
      acc += out;
    }
    __device__ __forceinline__ void finalize(float* out, int num_rows) {
      using BlockReduce = hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB>;
      __shared__ typename BlockReduce::TempStorage tmp_storage;
      acc = BlockReduce(tmp_storage).Sum(acc);
      if (threadIdx.x == 0) {
        for (int i = 0; i < NITEMS; ++i) {
          int row = blockIdx.x * NITEMS + i;
          if (row < num_rows)
            out[row * num_output_classes] = acc[i];
        }
      }
    }
};

template <int NITEMS, leaf_value_t leaf_payload_type, typename TOUTPUT, typename storage_type>
__global__ void infer_k(storage_type forest, predict_params params) {
  // cache the row for all threads to reuse
  extern __shared__ char smem[];
  float* sdata = (float*)smem;
  size_t rid = blockIdx.x * NITEMS;
  for (int j = 0; j < NITEMS; ++j) {
    for (int i = threadIdx.x; i < params.num_cols; i += blockDim.x) {
      size_t row = rid + j;
      sdata[j * params.num_cols + i] =
        row < params.num_rows ? params.data[row * params.num_cols + i] : 0.0f;
    }
  }
  __syncthreads();

  AggregateTrees<NITEMS, leaf_payload_type, TOUTPUT> acc(params.num_output_classes, nullptr);
  // one block works on NITEMS rows and the whole forest
  for (int j = threadIdx.x; j < forest.num_trees(); j += blockDim.x) {
    acc.accumulate(infer_one_tree<NITEMS, TOUTPUT>(forest[j], sdata, params.num_cols));
  }
  acc.finalize(params.preds, params.num_rows);
}

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream) {
  const int MAX_BATCH_ITEMS = 4;
  params.max_items =
    params.algo == algo_t::BATCH_TREE_REORG ? MAX_BATCH_ITEMS : 1;
  int num_items = params.max_shm / (sizeof(float) * params.num_cols);
  if (num_items == 0) {
    int max_cols = params.max_shm / sizeof(float);
    ASSERT(false, "p.num_cols == %d: too many features, only %d allowed",
           params.num_cols, max_cols);
  }
  num_items = std::min(num_items, params.max_items);
  int num_blocks = ceildiv(int(params.num_rows), num_items);
  int shm_sz = num_items * sizeof(float) * params.num_cols;
  switch (num_items) {
    case 1:
      switch (params.leaf_payload_type) {
        case FLOAT_SCALAR:
          ASSERT(params.num_output_classes <= 2, "wrong leaf payload for multi-class (>2) inference");
          infer_k<1, FLOAT_SCALAR, float><<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
          break;
        default:
          ASSERT(false, "only FLOAT_SCALAR supported as leaf_payload_type so far");
      }
      break;
    case 2:
      switch (params.leaf_payload_type) {
        case FLOAT_SCALAR:
          ASSERT(params.num_output_classes <= 2, "wrong leaf payload for multi-class (>2) inference");
          infer_k<2, FLOAT_SCALAR, float><<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
          break;
        default:
          ASSERT(false, "only FLOAT_SCALAR supported as leaf_payload_type so far");
      }
      break;
    case 3:
      switch (params.leaf_payload_type) {
        case FLOAT_SCALAR:
          ASSERT(params.num_output_classes <= 2, "wrong leaf payload for multi-class (>2) inference");
          infer_k<3, FLOAT_SCALAR, float><<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
          break;
        default:
          ASSERT(false, "only FLOAT_SCALAR supported as leaf_payload_type so far");
      }
      break;
    case 4:
      switch (params.leaf_payload_type) {
        case FLOAT_SCALAR:
          ASSERT(params.num_output_classes <= 2, "wrong leaf payload for multi-class (>2) inference");
          infer_k<4, FLOAT_SCALAR, float><<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
          break;
        default:
          ASSERT(false, "only FLOAT_SCALAR supported as leaf_payload_type so far");
      }
      break;
    default:
      ASSERT(false, "internal error: nitems > 4");
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template void infer<dense_storage>(dense_storage forest, predict_params params,
                                   hipStream_t stream);
template void infer<sparse_storage>(sparse_storage forest,
                                    predict_params params, hipStream_t stream);

}  // namespace fil
}  // namespace ML
