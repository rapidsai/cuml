#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cuml/fil/multi_sum.cuh>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;

// vec wraps float[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec {
  T data[N];
  explicit __host__ __device__ vec(T t = T()) {
#pragma unroll
    for (int i = 0; i < N; ++i) data[i] = t;
  }
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ void operator+=(vec<N, T>& a, const vec<N, T>& b) {
#pragma unroll
    for (int i = 0; i < N; ++i) a[i] += b[i];
  }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a,
                                                 const vec<N, T>& b) {
    vec<N, T> r = a;
    r += b;
    return r;
  }
};

typedef hipcub::KeyValuePair<int, float> best_margin_label;

template <int NITEMS>
__device__ __forceinline__ vec<NITEMS, best_margin_label> to_vec(
  int c, vec<NITEMS, float> margin) {
  vec<NITEMS, best_margin_label> ret;
#pragma unroll
  for (int i = 0; i < NITEMS; i++) ret[i] = best_margin_label(c, margin[i]);
  return ret;
}

struct ArgMax {
  template <int NITEMS>
  __host__ __device__ __forceinline__ vec<NITEMS, best_margin_label> operator()(
    vec<NITEMS, best_margin_label> a, vec<NITEMS, best_margin_label> b) const {
    vec<NITEMS, best_margin_label> c;
#pragma unroll
    for (int i = 0; i < NITEMS; i++) c[i] = hipcub::ArgMax()(a[i], b[i]);
    return c;
  }
};

/** tree_leaf_output returns the leaf outputs from the tree with leaf indices
    given by leaves for n_rows items. FULL_ITEMS indicates whether n_rows ==
    NITEMS, to allow the compiler to skip the conditional when unrolling the
    loop. */
template <typename output_type, bool FULL_NITEMS, int NITEMS,
          typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> tree_leaf_output(
  tree_type tree, int n_rows, int (&leaves)[NITEMS]) {
  vec<NITEMS, output_type> out(0);
#pragma unroll
  for (int j = 0; j < NITEMS; ++j) {
    if (FULL_NITEMS || j < n_rows) {
      /** dependent names are not considered templates by default, unless it's a
          member of a current [template] instantiation. As output<>() is a
          member function inherited from the base class, template
          output<output_type>() is required. */
      out[j] = tree[leaves[j]].template output<output_type>();
    }
  }
  return out;
}

template <int NITEMS, typename output_type, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> infer_one_tree(
  tree_type tree, const float* input, int cols, int n_rows) {
  // find the leaf nodes for each row
  int curr[NITEMS];
  // the first n_rows are active
  int mask = (1 << n_rows) - 1;
  for (int j = 0; j < NITEMS; ++j) curr[j] = 0;
  do {
#pragma unroll
    for (int j = 0; j < NITEMS; ++j) {
      auto n = tree[curr[j]];
      mask &= ~(n.is_leaf() << j);
      if ((mask & (1 << j)) != 0) {
        float val = input[j * cols + n.fid()];
        bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
        curr[j] = n.left(curr[j]) + cond;
      }
    }
  } while (mask != 0);

  // get the output from the leaves
  if (n_rows == NITEMS) {
    return tree_leaf_output<output_type, true>(tree, n_rows, curr);
  } else {
    return tree_leaf_output<output_type, false>(tree, n_rows, curr);
  }
}

template <typename output_type, typename tree_type>
__device__ __forceinline__ vec<1, output_type> infer_one_tree(
  tree_type tree, const float* input, int cols, int rows) {
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    float val = input[n.fid()];
    bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
    curr = n.left(curr) + cond;
  }
  vec<1, output_type> out;
  out[0] = tree[curr].base_node::output<output_type>();
  return out;
}

// the device template should achieve the best performance, using up-to-date
// CUB defaults
template <int NITEMS>
using BlockReduce = typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB>;
template <int NITEMS>
using BlockReduceBestClass =
  typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB>;
/**
The shared memory requirements for finalization stage may differ based
on the set of PTX architectures the kernels were compiled for, as well as 
the CUDA compute capability of the device chosen for computation.

TODO (levsnv): run a test kernel during forest init to determine the compute capability
chosen for the inference, for an accurate sizeof(BlockReduce::TempStorage),
which is used in determining max NITEMS or max input data columns.

600 is the __CUDA_ARCH__ for Pascal (6.0) GPUs, which is not defined in
host code.
6.0 is the earliest compute capability supported by FIL and RAPIDS in general.
See https://rapids.ai/start.html as well as cmake defaults.
*/
// values below are defaults as of this change.
template <int NITEMS>
using BlockReduceHost =
  typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB,
                            hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1, 600>;
template <int NITEMS>
using BlockReduceHostBestClass =
  typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB,
                            hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1, 600>;

template <int NITEMS,
          leaf_algo_t leaf_algo>  // = FLOAT_UNARY_BINARY
struct tree_aggregator_t {
  vec<NITEMS, float> acc;
  void* tmp_storage;

  /** shared memory footprint of the accumulator during
  the finalization of forest inference kernel, when infer_k output
  value is computed.
  num_classes is used for other template parameters */
  static size_t smem_finalize_footprint(int num_classes) {
    return sizeof(typename BlockReduceHost<NITEMS>::TempStorage);
  }

  /** shared memory footprint of the accumulator during
  the accumulation of forest inference, when individual trees
  are inferred and partial aggregates are accumulated.
  num_classes is used for other template parameters */
  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  /** 
  num_classes is used for other template parameters */
  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : tmp_storage(finalize_workspace) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int output_stride) {
    __syncthreads();
    typedef typename BlockReduce<NITEMS>::TempStorage TempStorage;
    acc = BlockReduce<NITEMS>(*(TempStorage*)tmp_storage).Sum(acc);
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < num_rows; ++row)
      out[row * output_stride] = acc[row];
  }
};

struct finalize_block {
  void* tmp_storage;
  int num_classes;

  __device__ __forceinline__ finalize_block(void* tmp_storage_,
                                            int num_classes_)
    : tmp_storage(tmp_storage_), num_classes(num_classes_) {}

  template <int NITEMS>
  static __host__ __device__ size_t smem_footprint() {
#ifdef __CUDA_ARCH__
    return sizeof(typename BlockReduceBestClass<NITEMS>::TempStorage);
#else
    return sizeof(typename BlockReduceHostBestClass<NITEMS>::TempStorage);
#endif
  }

  template <int NITEMS>
  __device__ __forceinline__ void write_best_class_in_block(
    vec<NITEMS, best_margin_label> best, int valid_threads, float* out,
    int num_rows) {
    // find best class per block (for each of the NITEMS rows)
    typedef BlockReduceBestClass<NITEMS> BlockReduceT;
    best = BlockReduceT(*(typename BlockReduceT::TempStorage*)tmp_storage)
             .Reduce(best, ArgMax(), valid_threads);
    // write it out to global memory
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < num_rows; ++row) out[row] = best[row].key;
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_FEW_CLASSES> : finalize_block {
  vec<NITEMS, float> acc;
  vec<NITEMS, float>* per_thread;

  static size_t smem_finalize_footprint(int num_classes) {
    size_t phase1 =
      (FIL_TPB - FIL_TPB % num_classes) * sizeof(vec<NITEMS, float>);
    size_t phase2 = finalize_block::smem_footprint<NITEMS>();
    return std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : finalize_block(finalize_workspace, params.num_classes),
      per_thread((vec<NITEMS, float>*)finalize_workspace) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  // block-reduce the best candidate class and write it out to global memory
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    __syncthreads();  // free up input row
    // load margin into shared memory
    per_thread[threadIdx.x] = acc;
    __syncthreads();
    acc = multi_sum<6>(per_thread, num_classes, blockDim.x / num_classes);
    __syncthreads();
    write_best_class_in_block(to_vec(threadIdx.x, acc), num_classes, out,
                              num_rows);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_MANY_CLASSES>
  : finalize_block {
  vec<NITEMS, float> acc;
  vec<NITEMS, float>* per_class_margin;

  static size_t smem_finalize_footprint(int num_classes) {
    // not accounting for lingering accumulate_footprint during finalize()
    return finalize_block::smem_footprint<NITEMS>();
  }

  static __host__ __device__ size_t smem_accumulate_footprint(int num_classes) {
    return num_classes * sizeof(vec<NITEMS, float>);
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : finalize_block(finalize_workspace, params.num_classes),
      per_class_margin((vec<NITEMS, float>*)accumulate_workspace) {
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      per_class_margin[c] = vec<NITEMS, float>();  // initialize to 0.0f
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    // since threads are assigned to consecutive classes, no need for atomics
    per_class_margin[tree % num_classes] += single_tree_prediction;
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    // reduce per-class candidate margins to one best class candidate
    // per thread (for each of the NITEMS rows)
    vec<NITEMS, best_margin_label> best({-1, -INFINITY});

    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      best = ArgMax()(best, to_vec(c, per_class_margin[c]));

    __syncthreads();  // free up per_class_margin[]
    write_best_class_in_block(best, blockDim.x, out, num_rows);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, CATEGORICAL_LEAF> {
  // could switch to unsigned short to save shared memory
  // provided raft::myAtomicAdd(short*) simulated with appropriate shifts
  int* votes;
  int num_classes;

  static size_t smem_finalize_footprint(int num_classes) {
    // not accounting for lingering accumulate_footprint during finalize()
    return 0;
  }
  static size_t smem_accumulate_footprint(int num_classes) {
    return sizeof(int) * num_classes * NITEMS;
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : num_classes(params.num_classes), votes((int*)accumulate_workspace) {
    for (int c = threadIdx.x; c < num_classes; c += FIL_TPB * NITEMS)
#pragma unroll
      for (int item = 0; item < NITEMS; ++item) votes[c * NITEMS + item] = 0;
    // __syncthreads() is called in infer_k
  }
  __device__ __forceinline__ void accumulate(
    vec<NITEMS, int> single_tree_prediction, int tree) {
#pragma unroll
    for (int item = 0; item < NITEMS; ++item)
      raft::myAtomicAdd(votes + single_tree_prediction[item] * NITEMS + item,
                        1);
  }
  // class probabilities or regression. for regression, num_classes
  // is just the number of outputs for each data instance
  __device__ __forceinline__ void finalize_multiple_outputs(float* out,
                                                            int num_rows) {
    __syncthreads();
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x) {
#pragma unroll
      for (int row = 0; row < num_rows; ++row)
        out[row * num_classes + c] = votes[c * NITEMS + row];
    }
  }
  // using this when predicting a single class label, as opposed to sparse class vector
  // or class probabilities or regression
  __device__ __forceinline__ void finalize_class_label(float* out,
                                                       int num_rows) {
    __syncthreads();
    int item = threadIdx.x;
    int row = item;
    if (item < NITEMS && row < num_rows) {
      int max_votes = 0;
      int best_class = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (votes[c * NITEMS + item] > max_votes) {
          max_votes = votes[c * NITEMS + item];
          best_class = c;
        }
      }
      out[row] = best_class;
    }
  }
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    if (num_outputs > 1) {
      // only supporting num_outputs == num_classes
      finalize_multiple_outputs(out, num_rows);
    } else {
      finalize_class_label(out, num_rows);
    }
  }
};

template <int NITEMS, leaf_algo_t leaf_algo, bool cols_in_shmem,
          class storage_type>
__global__ void infer_k(storage_type forest, predict_params params) {
  extern __shared__ char smem[];
  float* sdata = (float*)smem;
  int num_cols = params.num_cols;
  for (size_t block_row0 = blockIdx.x * NITEMS; block_row0 < params.num_rows;
       block_row0 += NITEMS * gridDim.x) {
    size_t num_input_rows = min((size_t)NITEMS, params.num_rows - block_row0);
    const float* block_input = params.data + block_row0 * num_cols;
    if (cols_in_shmem) {
      // cache the row for all threads to reuse
      size_t feature = 0;
#pragma unroll
      for (feature = threadIdx.x; feature < num_input_rows * num_cols;
           feature += blockDim.x)
        sdata[feature] = block_input[feature];
#pragma unroll
      for (; feature < NITEMS * num_cols; feature += blockDim.x)
        sdata[feature] = 0.0f;
    }

    tree_aggregator_t<NITEMS, leaf_algo> acc(
      params, (char*)sdata + params.cols_shmem_size(), sdata);

    __syncthreads();  // for both row cache init and acc init

    // one block works on NITEMS rows and the whole forest
    for (int j = threadIdx.x; j - threadIdx.x < forest.num_trees();
         j += blockDim.x) {
      /* j - threadIdx.x < forest.num_trees() is a necessary but block-uniform
         condition for "j < forest.num_trees()". It lets use __syncthreads()
         and is made exact below.
      */
      if (j < forest.num_trees()) {
        acc.accumulate(infer_one_tree<NITEMS, leaf_output_t<leaf_algo>::T>(
                         forest[j], cols_in_shmem ? sdata : block_input,
                         num_cols, num_input_rows),
                       j);
      }
      if (leaf_algo == GROVE_PER_CLASS_MANY_CLASSES) __syncthreads();
    }
    acc.finalize(params.preds + params.num_outputs * block_row0, num_input_rows,
                 params.num_outputs);
    __syncthreads();  // free up acc's shared memory resources for next row set
  }
}

template <int NITEMS, leaf_algo_t leaf_algo>
size_t shmem_size_params::get_smem_footprint() {
  size_t finalize_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_finalize_footprint(num_classes);
  size_t accumulate_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_accumulate_footprint(
      num_classes) +
    cols_shmem_size();

  return std::max(accumulate_footprint, finalize_footprint);
}

template <int NITEMS>
size_t shmem_size_params::get_smem_footprint() {
  switch (leaf_algo) {
    case FLOAT_UNARY_BINARY:
      return get_smem_footprint<NITEMS, FLOAT_UNARY_BINARY>();
    case CATEGORICAL_LEAF:
      return get_smem_footprint<NITEMS, CATEGORICAL_LEAF>();
    case GROVE_PER_CLASS:
      if (num_classes > FIL_TPB)
        return get_smem_footprint<NITEMS, GROVE_PER_CLASS_MANY_CLASSES>();
      return get_smem_footprint<NITEMS, GROVE_PER_CLASS_FEW_CLASSES>();
    default:
      ASSERT(false, "internal error: unexpected leaf_algo_t");
  }
}

void shmem_size_params::compute_smem_footprint() {
  switch (n_items) {
    case 1:
      shm_sz = get_smem_footprint<1>();
      break;
    case 2:
      shm_sz = get_smem_footprint<2>();
      break;
    case 3:
      shm_sz = get_smem_footprint<3>();
      break;
    case 4:
      shm_sz = get_smem_footprint<4>();
      break;
    default:
      ASSERT(false, "internal error: n_items > 4");
  }
}

template <leaf_algo_t leaf_algo, bool cols_in_shmem, typename storage_type>
void infer_k_nitems_launcher(storage_type forest, predict_params params,
                             hipStream_t stream, int block_dim_x) {
  switch (params.n_items) {
    case 1:
      infer_k<1, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    case 2:
      infer_k<2, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    case 3:
      infer_k<3, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    case 4:
      infer_k<4, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, block_dim_x, params.shm_sz, stream>>>(forest,
                                                                    params);
      break;
    default:
      ASSERT(false, "internal error: nitems > 4");
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template <leaf_algo_t leaf_algo, typename storage_type>
void infer_k_launcher(storage_type forest, predict_params params,
                      hipStream_t stream, int blockdim_x) {
  params.num_blocks = params.num_blocks != 0
                        ? params.num_blocks
                        : raft::ceildiv(int(params.num_rows), params.n_items);
  if (params.cols_in_shmem) {
    infer_k_nitems_launcher<leaf_algo, true>(forest, params, stream,
                                             blockdim_x);
  } else {
    infer_k_nitems_launcher<leaf_algo, false>(forest, params, stream,
                                              blockdim_x);
  }
}

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream) {
  switch (params.leaf_algo) {
    case FLOAT_UNARY_BINARY:
      infer_k_launcher<FLOAT_UNARY_BINARY>(forest, params, stream, FIL_TPB);
      break;
    case GROVE_PER_CLASS:
      if (params.num_classes > FIL_TPB) {
        params.leaf_algo = GROVE_PER_CLASS_MANY_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_MANY_CLASSES>(forest, params, stream,
                                                       FIL_TPB);
      } else {
        params.leaf_algo = GROVE_PER_CLASS_FEW_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_FEW_CLASSES>(
          forest, params, stream, FIL_TPB - FIL_TPB % params.num_classes);
      }
      break;
    case CATEGORICAL_LEAF:
      infer_k_launcher<CATEGORICAL_LEAF>(forest, params, stream, FIL_TPB);
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_algo");
  }
}

template void infer<dense_storage>(dense_storage forest, predict_params params,
                                   hipStream_t stream);
template void infer<sparse_storage16>(sparse_storage16 forest,
                                      predict_params params,
                                      hipStream_t stream);
template void infer<sparse_storage8>(sparse_storage8 forest,
                                     predict_params params,
                                     hipStream_t stream);

}  // namespace fil
}  // namespace ML
