#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cuml/fil/multi_sum.cuh>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;

// vec wraps float[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec {
  T data[N];
  explicit __host__ __device__ vec(T t = T()) {
#pragma unroll
    for (int i = 0; i < N; ++i) data[i] = t;
  }
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ void operator+=(vec<N, T>& a, const vec<N, T>& b) {
#pragma unroll
    for (int i = 0; i < N; ++i) a[i] += b[i];
  }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a,
                                                 const vec<N, T>& b) {
    vec<N, T> r = a;
    r += b;
    return r;
  }
};

typedef hipcub::KeyValuePair<int, float> best_margin_label;

template <int NITEMS>
__device__ __forceinline__ vec<NITEMS, best_margin_label> to_vec(
  int c, vec<NITEMS, float> margin) {
  vec<NITEMS, best_margin_label> ret;
#pragma unroll
  for (int i = 0; i < NITEMS; i++) ret[i] = best_margin_label(c, margin[i]);
  return ret;
}

struct ArgMax {
  template <int NITEMS>
  __host__ __device__ __forceinline__ vec<NITEMS, best_margin_label> operator()(
    vec<NITEMS, best_margin_label> a, vec<NITEMS, best_margin_label> b) const {
    vec<NITEMS, best_margin_label> c;
#pragma unroll
    for (int i = 0; i < NITEMS; i++) c[i] = hipcub::ArgMax()(a[i], b[i]);
    return c;
  }
};

template <int NITEMS, typename output_type, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> infer_one_tree(
  tree_type tree, const float* input, int cols) {
  int curr[NITEMS];
  int mask = (1 << NITEMS) - 1;  // all active
  for (int j = 0; j < NITEMS; ++j) curr[j] = 0;
  do {
#pragma unroll
    for (int j = 0; j < NITEMS; ++j) {
      auto n = tree[curr[j]];
      if (n.is_leaf()) {
        mask &= ~(1 << j);
        continue;
      }
      float val = input[j * cols + n.fid()];
      bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
      curr[j] = n.left(curr[j]) + cond;
    }
  } while (mask != 0);
  vec<NITEMS, output_type> out;
#pragma unroll
  for (int j = 0; j < NITEMS; ++j) {
    /** dependent names are not considered templates by default,
        unless it's a member of a current [template] instantiation.
        alternatively, could have used .base_node::output<... */
    out[j] = tree[curr[j]].template output<output_type>();
  }
  return out;
}

template <typename output_type, typename tree_type>
__device__ __forceinline__ vec<1, output_type> infer_one_tree(
  tree_type tree, const float* input, int cols) {
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    float val = input[n.fid()];
    bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
    curr = n.left(curr) + cond;
  }
  vec<1, output_type> out;
  out[0] = tree[curr].base_node::output<output_type>();
  return out;
}

// the device template should achieve the best performance, using up-to-date
// CUB defaults
template <int NITEMS>
using BlockReduce = typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB>;
template <int NITEMS>
using BlockReduceBestClass =
  typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB>;
/**
The shared memory requirements for finalization stage may differ based
on the set of PTX architectures the kernels were compiled for, as well as 
the CUDA compute capability of the device chosen for computation.

TODO (levsnv): run a test kernel during forest init to determine the compute capability
chosen for the inference, for an accurate sizeof(BlockReduce::TempStorage),
which is used in determining max NITEMS or max input data columns.

600 is the __CUDA_ARCH__ for Pascal (6.0) GPUs, which is not defined in
host code.
6.0 is the earliest compute capability supported by FIL and RAPIDS in general.
See https://rapids.ai/start.html as well as cmake defaults.
*/
// values below are defaults as of this change.
template <int NITEMS>
using BlockReduceHost =
  typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB,
                            hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1, 600>;
template <int NITEMS>
using BlockReduceHostBestClass =
  typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB,
                            hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1, 600>;

template <int NITEMS,
          leaf_algo_t leaf_algo>  // = FLOAT_UNARY_BINARY
struct tree_aggregator_t {
  vec<NITEMS, float> acc;
  void* tmp_storage;

  /** shared memory footprint of the accumulator during
  the finalization of forest inference kernel, when infer_k output
  value is computed.
  num_classes is used for other template parameters */
  static size_t smem_finalize_footprint(predict_params params) {
    return sizeof(typename BlockReduceHost<NITEMS>::TempStorage);
  }

  /** shared memory footprint of the accumulator during
  the accumulation of forest inference, when individual trees
  are inferred and partial aggregates are accumulated.
  num_classes is used for other template parameters */
  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  /** 
  num_classes is used for other template parameters */
  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : tmp_storage(finalize_workspace) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int output_stride) {
    __syncthreads();
    typedef typename BlockReduce<NITEMS>::TempStorage TempStorage;
    acc = BlockReduce<NITEMS>(*(TempStorage*)tmp_storage).Sum(acc);
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < num_rows; ++row)
      out[row * output_stride] = acc[row];
  }
};

struct finalize_block {
  void* tmp_storage;
  int num_classes;

  __device__ __forceinline__ finalize_block(void* tmp_storage_,
                                            int num_classes_)
    : tmp_storage(tmp_storage_), num_classes(num_classes_) {}

  template <int NITEMS>
  static __host__ __device__ size_t smem_footprint() {
#ifdef __CUDA_ARCH__
    return sizeof(typename BlockReduceBestClass<NITEMS>::TempStorage);
#else
    return sizeof(typename BlockReduceHostBestClass<NITEMS>::TempStorage);
#endif
  }

  template <int NITEMS>
  __device__ __forceinline__ void write_best_class_in_block(
    vec<NITEMS, best_margin_label> best, int valid_threads, float* out,
    int num_rows) {
    // find best class per block (for each of the NITEMS rows)
    typedef BlockReduceBestClass<NITEMS> BlockReduceT;
    best = BlockReduceT(*(typename BlockReduceT::TempStorage*)tmp_storage)
             .Reduce(best, ArgMax(), valid_threads);
    // write it out to global memory
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < num_rows; ++row) out[row] = best[row].key;
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_FEW_CLASSES> : finalize_block {
  vec<NITEMS, float> acc;

  static size_t smem_finalize_footprint(predict_params params) {
    size_t phase1 =
      (FIL_TPB - FIL_TPB % params.num_classes) * sizeof(vec<NITEMS, float>);
    size_t phase2 = finalize_block::smem_footprint<NITEMS>();
    return std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : finalize_block(finalize_workspace, params.num_classes) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  // block-reduce the best candidate class and write it out to global memory
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    __syncthreads();  // free up input row
    // load margin into shared memory
    auto per_thread = (vec<NITEMS, float>*)tmp_storage;
    per_thread[threadIdx.x] = acc;
    __syncthreads();
    acc = multi_sum<6>(per_thread, num_classes, blockDim.x / num_classes);
    __syncthreads();
    write_best_class_in_block(to_vec(threadIdx.x, acc), num_classes, out,
                              num_rows);
  }
};

template <int NITEMS>
__host__ __device__ size_t cols_shmem_size(predict_params params) {
  return params.cols_in_shmem ? params.num_cols * NITEMS * sizeof(float) : 0;
}

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_MANY_CLASSES>
  : finalize_block {
  vec<NITEMS, float> acc;
  vec<NITEMS, float>* per_class_margin;

  static size_t smem_finalize_footprint(predict_params params) {
    size_t phase1 = cols_shmem_size<NITEMS>(params) +
                    smem_accumulate_footprint(params.num_classes);
    size_t phase2 =
      finalize_block::smem_footprint<NITEMS>() > cols_shmem_size<NITEMS>(params)
        ? cols_shmem_size<NITEMS>(params) +
            finalize_block::smem_footprint<NITEMS>()
        : finalize_block::smem_footprint<NITEMS>();
    return std::max(phase1, phase2);
  }

  static __host__ __device__ size_t smem_accumulate_footprint(int num_classes) {
    return num_classes * sizeof(vec<NITEMS, float>);
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    // if finalize_block fits into cols_shmem_size, overlap, since one is used
    // only during "finalize" stage, and another - only during "accumulate" stage.
    : finalize_block(finalize_block::smem_footprint<NITEMS>() >
                         cols_shmem_size<NITEMS>(params)
                       ? accumulate_workspace
                       : finalize_workspace,
                     params.num_classes),
      per_class_margin((vec<NITEMS, float>*)accumulate_workspace) {
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      per_class_margin[c] = vec<NITEMS, float>();  // initialize to 0.0f
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    // since threads are assigned to consecutive classes, no need for atomics
    per_class_margin[tree % num_classes] += single_tree_prediction;
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    // reduce per-class candidate margins to one best class candidate
    // per thread (for each of the NITEMS rows)
    vec<NITEMS, best_margin_label> best({-1, -INFINITY});

    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      best = ArgMax()(best, to_vec(c, per_class_margin[c]));

    __syncthreads();  // free up per_class_margin[]
    write_best_class_in_block(best, blockDim.x, out, num_rows);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, CATEGORICAL_LEAF> {
  // could switch to unsigned short to save shared memory
  // provided raft::myAtomicAdd(short*) simulated with appropriate shifts
  int* votes;
  int num_classes;

  static size_t smem_finalize_footprint(predict_params params) {
    return smem_accumulate_footprint(params.num_classes) +
           cols_shmem_size<NITEMS>(params);
  }
  static size_t smem_accumulate_footprint(int num_classes) {
    return sizeof(int) * num_classes * NITEMS;
  }

  __device__ __forceinline__ tree_aggregator_t(predict_params params,
                                               void* accumulate_workspace,
                                               void* finalize_workspace)
    : num_classes(params.num_classes), votes((int*)accumulate_workspace) {
    for (int c = threadIdx.x; c < num_classes; c += FIL_TPB * NITEMS)
#pragma unroll
      for (int item = 0; item < NITEMS; ++item) votes[c * NITEMS + item] = 0;
    // __syncthreads() is called in infer_k
  }
  __device__ __forceinline__ void accumulate(
    vec<NITEMS, int> single_tree_prediction, int tree) {
#pragma unroll
    for (int item = 0; item < NITEMS; ++item)
      raft::myAtomicAdd(votes + single_tree_prediction[item] * NITEMS + item,
                        1);
  }
  // class probabilities or regression. for regression, num_classes
  // is just the number of outputs for each data instance
  __device__ __forceinline__ void finalize_multiple_outputs(float* out,
                                                            int num_rows) {
    __syncthreads();
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x) {
#pragma unroll
      for (int row = 0; row < num_rows; ++row)
        out[row * num_classes + c] = votes[c * NITEMS + row];
    }
  }
  // using this when predicting a single class label, as opposed to sparse class vector
  // or class probabilities or regression
  __device__ __forceinline__ void finalize_class_label(float* out,
                                                       int num_rows) {
    __syncthreads();
    int item = threadIdx.x;
    int row = item;
    if (item < NITEMS && row < num_rows) {
      int max_votes = 0;
      int best_class = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (votes[c * NITEMS + item] > max_votes) {
          max_votes = votes[c * NITEMS + item];
          best_class = c;
        }
      }
      out[row] = best_class;
    }
  }
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    if (num_outputs > 1) {
      // only supporting num_outputs == num_classes
      finalize_multiple_outputs(out, num_rows);
    } else {
      finalize_class_label(out, num_rows);
    }
  }
};

template <int NITEMS, leaf_algo_t leaf_algo, bool cols_in_shmem,
          class storage_type>
__global__ void infer_k(storage_type forest, predict_params params) {
  extern __shared__ char smem[];
  float* sdata = (float*)smem;
  for (size_t block_row0 = blockIdx.x * NITEMS; block_row0 < params.num_rows;
       block_row0 += NITEMS * gridDim.x) {
    if (cols_in_shmem) {
      // cache the row for all threads to reuse
      for (size_t j = 0; j < NITEMS; ++j) {
        size_t row = block_row0 + j;
#pragma unroll
        for (int col = threadIdx.x; col < params.num_cols; col += blockDim.x) {
          sdata[j * params.num_cols + col] =
            row < params.num_rows ? params.data[row * params.num_cols + col]
                                  : 0.0f;
        }
      }
    }

    tree_aggregator_t<NITEMS, leaf_algo> acc(
      params, (char*)sdata + cols_shmem_size<NITEMS>(params), sdata);

    __syncthreads();  // for both row cache init and acc init

    // one block works on NITEMS rows and the whole forest
    for (int j = threadIdx.x; j - threadIdx.x < forest.num_trees();
         j += blockDim.x) {
      /* j - threadIdx.x < forest.num_trees() is a necessary but block-uniform
         condition for "j < forest.num_trees()". It lets use __syncthreads()
         and is made exact below.
      */
      if (j < forest.num_trees()) {
        acc.accumulate(
          infer_one_tree<NITEMS, leaf_output_t<leaf_algo>::T>(
            forest[j], cols_in_shmem ? sdata : params.data, params.num_cols),
          j);
      }
      if (leaf_algo == GROVE_PER_CLASS_MANY_CLASSES) __syncthreads();
    }
    acc.finalize(params.preds + params.num_outputs * block_row0,
                 min((size_t)NITEMS, params.num_rows - block_row0),
                 params.num_outputs);
    __syncthreads();  // free up acc's shared memory resources for next row set
  }
}

template <int NITEMS, leaf_algo_t leaf_algo>
size_t get_smem_footprint(predict_params params) {
  size_t finalize_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_finalize_footprint(params);
  size_t accumulate_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_accumulate_footprint(
      params.num_classes) +
    cols_shmem_size<NITEMS>(params);

  return std::max(accumulate_footprint, finalize_footprint);
}

template <leaf_algo_t leaf_algo, int NITEMS>
void try_nitems(int* num_items, size_t* shm_sz, predict_params params) {
  size_t peak_footprint = get_smem_footprint<NITEMS, leaf_algo>(params);
  if (peak_footprint <= params.max_shm) {
    *num_items = NITEMS;
    *shm_sz = peak_footprint;
  }
}

template <leaf_algo_t leaf_algo, bool cols_in_shmem, typename storage_type>
void infer_k_nitems_launcher(storage_type forest, predict_params params,
                             hipStream_t stream, int blockdim_x, int num_items,
                             size_t shm_sz) {
  switch (num_items) {
    case 1:
      infer_k<1, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 2:
      infer_k<2, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 3:
      infer_k<3, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 4:
      infer_k<4, leaf_algo, cols_in_shmem>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    default:
      ASSERT(false, "internal error: nitems > 4");
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template <leaf_algo_t leaf_algo, typename storage_type>
void infer_k_launcher(storage_type forest, predict_params params,
                      hipStream_t stream, int blockdim_x) {
  // TODO(levsnv): move this to forest init()
  /** searching for the most items per block while respecting the shared
  * memory limits creates a full linear programming problem.
  * solving it in a single equation looks less tractable than this */
  params.cols_in_shmem = true;
  int num_items = 0;
  size_t shm_sz;
  try_nitems<leaf_algo, 1>(&num_items, &shm_sz, params);
  if (num_items == 0) {
    params.cols_in_shmem = false;
    try_nitems<leaf_algo, 1>(&num_items, &shm_sz, params);
    try_nitems<leaf_algo, 2>(&num_items, &shm_sz, params);
    try_nitems<leaf_algo, 3>(&num_items, &shm_sz, params);
    try_nitems<leaf_algo, 4>(&num_items, &shm_sz, params);
    ASSERT(num_items != 0, "FIL out of shared memory. >>5'000 classes?");
  } else if (params.algo == algo_t::BATCH_TREE_REORG) {
    try_nitems<leaf_algo, 2>(&num_items, &shm_sz, params);
    try_nitems<leaf_algo, 3>(&num_items, &shm_sz, params);
    try_nitems<leaf_algo, 4>(&num_items, &shm_sz, params);
  }
  params.num_blocks = params.num_blocks != 0
                        ? params.num_blocks
                        : raft::ceildiv(int(params.num_rows), num_items);
  if (params.cols_in_shmem) {
    infer_k_nitems_launcher<leaf_algo, true>(forest, params, stream, blockdim_x,
                                             num_items, shm_sz);
  } else {
    infer_k_nitems_launcher<leaf_algo, false>(forest, params, stream,
                                              blockdim_x, num_items, shm_sz);
  }
}

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream) {
  switch (params.leaf_algo) {
    case FLOAT_UNARY_BINARY:
      infer_k_launcher<FLOAT_UNARY_BINARY>(forest, params, stream, FIL_TPB);
      break;
    case GROVE_PER_CLASS:
      if (params.num_classes > FIL_TPB) {
        params.leaf_algo = GROVE_PER_CLASS_MANY_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_MANY_CLASSES>(forest, params, stream,
                                                       FIL_TPB);
      } else {
        params.leaf_algo = GROVE_PER_CLASS_FEW_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_FEW_CLASSES>(
          forest, params, stream, FIL_TPB - FIL_TPB % params.num_classes);
      }
      break;
    case CATEGORICAL_LEAF:
      infer_k_launcher<CATEGORICAL_LEAF>(forest, params, stream, FIL_TPB);
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_algo");
  }
}

template void infer<dense_storage>(dense_storage forest, predict_params params,
                                   hipStream_t stream);
template void infer<sparse_storage16>(sparse_storage16 forest,
                                      predict_params params,
                                      hipStream_t stream);
template void infer<sparse_storage8>(sparse_storage8 forest,
                                     predict_params params,
                                     hipStream_t stream);

}  // namespace fil
}  // namespace ML
