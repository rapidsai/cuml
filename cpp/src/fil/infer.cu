#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cuml/fil/multi_sum.cuh>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;

// vec wraps float[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec {
  T data[N];
  explicit __host__ __device__ vec(T t = T()) {
#pragma unroll
    for (int i = 0; i < N; ++i) data[i] = t;
  }
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ void operator+=(vec<N, T>& a, const vec<N, T>& b) {
#pragma unroll
    for (int i = 0; i < N; ++i) a[i] += b[i];
  }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a,
                                                 const vec<N, T>& b) {
    vec<N, T> r = a;
    r += b;
    return r;
  }
};

typedef hipcub::KeyValuePair<int, float> best_margin_label;

template <int NITEMS>
__device__ __forceinline__ vec<NITEMS, best_margin_label> to_vec(
  int c, vec<NITEMS, float> margin) {
  vec<NITEMS, best_margin_label> ret;
#pragma unroll
  for (int i = 0; i < NITEMS; i++) ret[i] = best_margin_label(c, margin[i]);
  return ret;
}

struct ArgMax {
  template <int NITEMS>
  __host__ __device__ __forceinline__ vec<NITEMS, best_margin_label> operator()(
    vec<NITEMS, best_margin_label> a, vec<NITEMS, best_margin_label> b) const {
    vec<NITEMS, best_margin_label> c;
#pragma unroll
    for (int i = 0; i < NITEMS; i++) c[i] = hipcub::ArgMax()(a[i], b[i]);
    return c;
  }
};

template <int NITEMS, typename output_type, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> infer_one_tree(
  tree_type tree, float* sdata, int cols) {
  int curr[NITEMS];
  int mask = (1 << NITEMS) - 1;  // all active
  for (int j = 0; j < NITEMS; ++j) curr[j] = 0;
  do {
#pragma unroll
    for (int j = 0; j < NITEMS; ++j) {
      auto n = tree[curr[j]];
      if (n.is_leaf()) {
        mask &= ~(1 << j);
        continue;
      }
      float val = sdata[j * cols + n.fid()];
      bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
      curr[j] = n.left(curr[j]) + cond;
    }
  } while (mask != 0);
  vec<NITEMS, output_type> out;
#pragma unroll
  for (int j = 0; j < NITEMS; ++j) {
    /** dependent names are not considered templates by default,
        unless it's a member of a current [template] instantiation.
        alternatively, could have used .base_node::output<... */
    out[j] = tree[curr[j]].template output<output_type>();
  }
  return out;
}

template <typename output_type, typename tree_type>
__device__ __forceinline__ vec<1, output_type> infer_one_tree(tree_type tree,
                                                              float* sdata,
                                                              int cols) {
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    float val = sdata[n.fid()];
    bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
    curr = n.left(curr) + cond;
  }
  vec<1, output_type> out;
  out[0] = tree[curr].base_node::output<output_type>();
  return out;
}

// the device template should achieve the best performance, using up-to-date
// CUB defaults
template <int NITEMS>
using BlockReduce = typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB>;
template <int NITEMS>
using BlockReduceBestClass =
  typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB>;
/**
The shared memory requirements for finalization stage may differ based
on the set of PTX architectures the kernels were compiled for, as well as 
the CUDA compute capability of the device chosen for computation.

TODO (levsnv): run a test kernel during forest init to determine the compute capability
chosen for the inference, for an accurate sizeof(BlockReduce::TempStorage),
which is used in determining max NITEMS or max input data columns.

600 is the __CUDA_ARCH__ for Pascal (6.0) GPUs, which is not defined in
host code.
6.0 is the earliest compute capability supported by FIL and RAPIDS in general.
See https://rapids.ai/start.html as well as cmake defaults.
*/
// values below are defaults as of this change.
template <int NITEMS>
using BlockReduceHost =
  typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB,
                            hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1, 600>;
template <int NITEMS>
using BlockReduceHostBestClass =
  typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB,
                            hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1, 600>;

template <int NITEMS,
          leaf_algo_t leaf_algo>  // = FLOAT_UNARY_BINARY
struct tree_aggregator_t {
  vec<NITEMS, float> acc;
  void* tmp_storage;

  /** shared memory footprint of the accumulator during
  the finalization of forest inference kernel, when infer_k output
  value is computed.
  num_classes is used for other template parameters */
  static size_t smem_finalize_footprint(int num_classes) {
    return sizeof(typename BlockReduceHost<NITEMS>::TempStorage);
  }

  /** shared memory footprint of the accumulator during
  the accumulation of forest inference, when individual trees
  are inferred and partial aggregates are accumulated.
  num_classes is used for other template parameters */
  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  /** 
  num_classes is used for other template parameters */
  __device__ __forceinline__ tree_aggregator_t(int num_classes,
                                               void* shared_workspace, size_t)
    : tmp_storage(shared_workspace) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int output_stride) {
    __syncthreads();
    typedef typename BlockReduce<NITEMS>::TempStorage TempStorage;
    acc = BlockReduce<NITEMS>(*(TempStorage*)tmp_storage).Sum(acc);
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < num_rows; ++row)
      out[row * output_stride] = acc[row];
  }
};

struct finalize_block {
  void* tmp_storage;
  int num_classes;

  __device__ __forceinline__ finalize_block(void* tmp_storage_,
                                            int num_classes_)
    : tmp_storage(tmp_storage_), num_classes(num_classes_) {}

  template <int NITEMS>
  static size_t smem_footprint() {
    return sizeof(typename BlockReduceHostBestClass<NITEMS>::TempStorage);
  }

  template <int NITEMS>
  __device__ __forceinline__ void write_best_class_in_block(
    vec<NITEMS, best_margin_label> best, int valid_threads, float* out,
    int num_rows) {
    // find best class per block (for each of the NITEMS rows)
    typedef BlockReduceBestClass<NITEMS> BlockReduceT;
    best = BlockReduceT(*(typename BlockReduceT::TempStorage*)tmp_storage)
             .Reduce(best, ArgMax(), valid_threads);
    // write it out to global memory
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < num_rows; ++row) out[row] = best[row].key;
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_FEW_CLASSES> : finalize_block {
  vec<NITEMS, float> acc;

  static size_t smem_finalize_footprint(int num_classes) {
    size_t phase1 =
      (FIL_TPB - FIL_TPB % num_classes) * sizeof(vec<NITEMS, float>);
    size_t phase2 = finalize_block::smem_footprint<NITEMS>();
    return std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  __device__ __forceinline__ tree_aggregator_t(int num_classes_,
                                               void* shared_workspace, size_t)
    : finalize_block(shared_workspace, num_classes_) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  // block-reduce the best candidate class and write it out to global memory
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    __syncthreads();  // free up input row
    // load margin into shared memory
    auto per_thread = (vec<NITEMS, float>*)tmp_storage;
    per_thread[threadIdx.x] = acc;
    __syncthreads();
    acc = multi_sum<6>(per_thread, num_classes, blockDim.x / num_classes);
    __syncthreads();
    write_best_class_in_block(to_vec(threadIdx.x, acc), num_classes, out,
                              num_rows);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_MANY_CLASSES>
  : finalize_block {
  vec<NITEMS, float> acc;
  vec<NITEMS, float>* per_class_margin;

  static size_t smem_finalize_footprint(int num_classes) {
    size_t phase1 = num_classes * sizeof(vec<NITEMS, float>);
    size_t phase2 = finalize_block::smem_footprint<NITEMS>();
    return std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) {
    return num_classes * sizeof(vec<NITEMS, float>);
  }

  __device__ __forceinline__ tree_aggregator_t(int num_classes_,
                                               void* shared_workspace,
                                               size_t data_row_size)
    : finalize_block(shared_workspace, num_classes_),
      per_class_margin(
        (vec<NITEMS, float>*)((char*)shared_workspace + data_row_size)) {
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      per_class_margin[c] = vec<NITEMS, float>();  // initialize to 0.0f
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    // since threads are assigned to consecutive classes, no need for atomics
    per_class_margin[tree % num_classes] += single_tree_prediction;
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    // reduce per-class candidate margins to one best class candidate
    // per thread (for each of the NITEMS rows)
    vec<NITEMS, best_margin_label> best({-1, -INFINITY});

    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      best = ArgMax()(best, to_vec(c, per_class_margin[c]));

    __syncthreads();  // free up per_class_margin[]
    write_best_class_in_block(best, blockDim.x, out, num_rows);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, CATEGORICAL_LEAF> {
  // could switch to unsigned short to save shared memory
  // provided raft::myAtomicAdd(short*) simulated with appropriate shifts
  int* votes;
  int num_classes;

  static size_t smem_finalize_footprint(int num_classes) {
    return sizeof(int) * num_classes * NITEMS;
  }
  static size_t smem_accumulate_footprint(int num_classes) {
    return smem_finalize_footprint(num_classes);
  }

  __device__ __forceinline__ tree_aggregator_t(int num_classes_,
                                               void* shared_workspace,
                                               size_t data_row_size)
    : num_classes(num_classes_),
      votes((int*)(data_row_size + (char*)shared_workspace)) {
    for (int c = threadIdx.x; c < num_classes; c += FIL_TPB * NITEMS)
#pragma unroll
      for (int item = 0; item < NITEMS; ++item) votes[c * NITEMS + item] = 0;
    // __syncthreads() is called in infer_k
  }
  __device__ __forceinline__ void accumulate(
    vec<NITEMS, int> single_tree_prediction, int tree) {
#pragma unroll
    for (int item = 0; item < NITEMS; ++item)
      raft::myAtomicAdd(votes + single_tree_prediction[item] * NITEMS + item,
                        1);
  }
  // class probabilities or regression. for regression, num_classes
  // is just the number of outputs for each data instance
  __device__ __forceinline__ void finalize_multiple_outputs(float* out,
                                                            int num_rows) {
    __syncthreads();
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x) {
#pragma unroll
      for (int row = 0; row < num_rows; ++row)
        out[row * num_classes + c] = votes[c * NITEMS + row];
    }
  }
  // using this when predicting a single class label, as opposed to sparse class vector
  // or class probabilities or regression
  __device__ __forceinline__ void finalize_class_label(float* out,
                                                       int num_rows) {
    __syncthreads();
    int item = threadIdx.x;
    int row = item;
    if (item < NITEMS && row < num_rows) {
      int max_votes = 0;
      int best_class = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (votes[c * NITEMS + item] > max_votes) {
          max_votes = votes[c * NITEMS + item];
          best_class = c;
        }
      }
      out[row] = best_class;
    }
  }
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs) {
    if (num_outputs > 1) {
      // only supporting num_outputs == num_classes
      finalize_multiple_outputs(out, num_rows);
    } else {
      finalize_class_label(out, num_rows);
    }
  }
};

template <int NITEMS, leaf_algo_t leaf_algo, class storage_type>
__global__ void infer_k(storage_type forest, predict_params params) {
  extern __shared__ char smem[];
  float* sdata = (float*)smem;
  for (size_t block_row0 = blockIdx.x * NITEMS; block_row0 < params.num_rows;
       block_row0 += NITEMS * gridDim.x) {
    // cache the row for all threads to reuse
    for (size_t j = 0; j < NITEMS; ++j) {
      size_t row = block_row0 + j;
#pragma unroll
      for (int col = threadIdx.x; col < params.num_cols; col += blockDim.x) {
        sdata[j * params.num_cols + col] =
          row < params.num_rows ? params.data[row * params.num_cols + col]
                                : 0.0f;
      }
    }

    tree_aggregator_t<NITEMS, leaf_algo> acc(
      params.num_classes, sdata, params.num_cols * NITEMS * sizeof(float));

    __syncthreads();  // for both row cache init and acc init

    // one block works on NITEMS rows and the whole forest
    for (int j = threadIdx.x; j - threadIdx.x < forest.num_trees();
         j += blockDim.x) {
      /* j - threadIdx.x < forest.num_trees() is a necessary but block-uniform
         condition for "j < forest.num_trees()". It lets use __syncthreads()
         and is made exact below.
      */
      if (j < forest.num_trees()) {
        acc.accumulate(infer_one_tree<NITEMS, leaf_output_t<leaf_algo>::T>(
                         forest[j], sdata, params.num_cols),
                       j);
      }
      if (leaf_algo == GROVE_PER_CLASS_MANY_CLASSES) __syncthreads();
    }
    acc.finalize(params.preds + params.num_outputs * block_row0,
                 min((size_t)NITEMS, params.num_rows - block_row0),
                 params.num_outputs);
    __syncthreads();  // free up acc's shared memory resources for next row set
  }
}

template <int NITEMS, leaf_algo_t leaf_algo>
size_t get_smem_footprint(predict_params params) {
  size_t finalize_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_finalize_footprint(
      params.num_classes);
  size_t accumulate_footprint =
    sizeof(float) * params.num_cols * NITEMS +
    tree_aggregator_t<NITEMS, leaf_algo>::smem_accumulate_footprint(
      params.num_classes);

  return std::max(accumulate_footprint, finalize_footprint);
}

template <leaf_algo_t leaf_algo, typename storage_type>
void infer_k_launcher(storage_type forest, predict_params params,
                      hipStream_t stream, int blockdim_x) {
  const int MAX_BATCH_ITEMS = 4;
  params.max_items =
    params.algo == algo_t::BATCH_TREE_REORG ? MAX_BATCH_ITEMS : 1;

  /** searching for the most items per block while respecting the shared
  * memory limits creates a full linear programming problem.
  * solving it in a single equation looks less tractable than this */
  int num_items = 0;
  size_t shm_sz = 0;
  for (int nitems = 1; nitems <= params.max_items; ++nitems) {
    size_t peak_footprint;
    switch (nitems) {
      case 1:
        peak_footprint = get_smem_footprint<1, leaf_algo>(params);
        break;
      case 2:
        peak_footprint = get_smem_footprint<2, leaf_algo>(params);
        break;
      case 3:
        peak_footprint = get_smem_footprint<3, leaf_algo>(params);
        break;
      case 4:
        peak_footprint = get_smem_footprint<4, leaf_algo>(params);
        break;
      default:
        ASSERT(false, "internal error: nitems > 4");
    }
    // for data row
    if (peak_footprint <= params.max_shm) {
      num_items = nitems;
      shm_sz = peak_footprint;
    }
  }
  if (num_items == 0) {
    int given_num_cols = params.num_cols;
    // starting with maximum that might fit in shared memory, in case
    // given_num_cols is a random large int
    params.num_cols = params.max_shm / sizeof(float);
    // since we're crashing, this will not take too long
    while (params.num_cols > 0 &&
           get_smem_footprint<1, leaf_algo>(params) > params.max_shm) {
      --params.num_cols;
    }
    ASSERT(false, "p.num_cols == %d: too many features, only %d allowed",
           given_num_cols, params.num_cols);
  }
  params.num_blocks = params.num_blocks != 0
                        ? params.num_blocks
                        : raft::ceildiv(int(params.num_rows), num_items);
  switch (num_items) {
    case 1:
      infer_k<1, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 2:
      infer_k<2, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 3:
      infer_k<3, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 4:
      infer_k<4, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    default:
      ASSERT(false, "internal error: nitems > 4");
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream) {
  switch (params.leaf_algo) {
    case FLOAT_UNARY_BINARY:
      infer_k_launcher<FLOAT_UNARY_BINARY>(forest, params, stream, FIL_TPB);
      break;
    case GROVE_PER_CLASS:
      if (params.num_classes > FIL_TPB) {
        params.leaf_algo = GROVE_PER_CLASS_MANY_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_MANY_CLASSES>(forest, params, stream,
                                                       FIL_TPB);
      } else {
        params.leaf_algo = GROVE_PER_CLASS_FEW_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_FEW_CLASSES>(
          forest, params, stream, FIL_TPB - FIL_TPB % params.num_classes);
      }
      break;
    case CATEGORICAL_LEAF:
      infer_k_launcher<CATEGORICAL_LEAF>(forest, params, stream, FIL_TPB);
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_algo");
  }
}

template void infer<dense_storage>(dense_storage forest, predict_params params,
                                   hipStream_t stream);
template void infer<sparse_storage16>(sparse_storage16 forest,
                                      predict_params params,
                                      hipStream_t stream);
template void infer<sparse_storage8>(sparse_storage8 forest,
                                     predict_params params,
                                     hipStream_t stream);

}  // namespace fil
}  // namespace ML
