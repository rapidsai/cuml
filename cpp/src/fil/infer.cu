#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;

// vec wraps float[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec {
  T data[N];
  inline vec() = default;  // zeros for numerical member vars
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ void operator+=(vec<N, T>& a, const vec<N, T>& b) {
#pragma unroll
    for (int i = 0; i < N; ++i) a[i] += b[i];
  }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a,
                                                 const vec<N, T>& b) {
    vec<N, T> r = a;
    r += b;
    return r;
  }
};

#define __forceinline__ 

template <int NITEMS, typename output_type, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> infer_one_tree(
  tree_type tree, float* sdata, int cols) {
  int curr[NITEMS];
  int mask = (1 << NITEMS) - 1;  // all active
  for (int j = 0; j < NITEMS; ++j) curr[j] = 0;
  do {
#pragma unroll
    for (int j = 0; j < NITEMS; ++j) {
      if ((mask >> j) & 1 == 0) continue;
      auto n = tree[curr[j]];
      if (n.is_leaf()) {
        mask &= ~(1 << j);
        continue;
      }
      float val = sdata[j * cols + n.fid()];
      bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
      curr[j] = n.left(curr[j]) + cond;
    }
  } while (mask != 0);
  vec<NITEMS, output_type> out;
#pragma unroll
  for (int j = 0; j < NITEMS; ++j) out[j] = tree[curr[j]].output();
  return out;
}

template <typename output_type, typename tree_type>
__device__ __forceinline__ vec<1, output_type> infer_one_tree(tree_type tree,
                                                              float* sdata,
                                                              int cols) {
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    float val = sdata[n.fid()];
    bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
    curr = n.left(curr) + cond;
  }
  vec<1, output_type> out;
  out[0] = tree[curr].output();
  return out;
}

template <int NITEMS,
          leaf_value_desc_t leaf_payload_type,  // = FLOAT_SCALAR,
          typename node_payload_type>           // = float>
struct tree_aggregator_t {
  vec<NITEMS, float> acc;
  int num_output_classes;

  __device__ __forceinline__ tree_aggregator_t(int num_output_classes_, void*)
    : num_output_classes(num_output_classes_) {
    // TODO: even if num_output_classes == 2, in regression, this needs to change
  }
  __device__ __forceinline__ void accumulate(vec<NITEMS, val_t> out) {
#pragma unroll
    for (int i = 0; i < NITEMS; ++i) acc[i] += out[i].f;
  }
  __device__ __forceinline__ void finalize(float* out, int num_rows) {
    using BlockReduce = hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB>;
    __shared__ typename BlockReduce::TempStorage tmp_storage;
    acc = BlockReduce(tmp_storage).Sum(acc);
    if (threadIdx.x == 0) {
      for (int i = 0; i < NITEMS; ++i) {
        int row = blockIdx.x * NITEMS + i;
        if (row < num_rows) out[row * num_output_classes] = acc[i];
        //TODO for 2 output values, will need to change the above line
        // to fix regression
      }
    }
  }
  __device__ __forceinline__ void finalize_class_label(float* out,
                                                       int num_rows) {
    finalize(out, num_rows);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, INT_CLASS_LABEL, unsigned int> {
  typedef unsigned int class_label_t;
  typedef unsigned int vote_count_t;
  // can switch to unsigned short to save shared memory
  // provided atomicInc(short*) simulated with atomicAdd with appropriate shifts
  vote_count_t* votes;
  int num_output_classes;

  __device__ __forceinline__ tree_aggregator_t(int num_output_classes_,
                                               void* shared_workspace)
    : votes((vote_count_t*)shared_workspace),
      num_output_classes(num_output_classes_) {
    for (int c = threadIdx.x; c < num_output_classes; c += FIL_TPB * NITEMS)
#pragma unroll
      for (int i = 0; i < NITEMS; ++i) votes[c * NITEMS + i] = 0;
    //__syncthreads(); // happening outside
  }
  __device__ __forceinline__ void accumulate(vec<NITEMS, val_t> out) {
#pragma unroll
    for (int i = 0; i < NITEMS; ++i)
      atomicAdd(votes + out[i].idx * NITEMS + i, 1);
  }
  __device__ __forceinline__ void finalize(float* out, int num_rows) {
    __syncthreads();
    int item = threadIdx.x;
    int row = blockIdx.x * NITEMS + item;
    if ((item < NITEMS) && (row < num_rows)) {
#pragma unroll
      for (int c = 0; c < num_output_classes; ++c)
        out[row * num_output_classes + c] = votes[c * NITEMS + item];
    }
  }
  // using this when predicting a single class label, as opposed to sparse class vector
  // or class probabilities or regression
  __device__ __forceinline__ void finalize_class_label(float* out,
                                                       int num_rows) {
    __syncthreads();
    int item = threadIdx.x;
    int row = blockIdx.x * NITEMS + item;
    if ((item < NITEMS) && (row < num_rows)) {
      vote_count_t max_votes = 0;
      class_label_t best_class = 0;
      for (int c = 0; c < num_output_classes; ++c)
        if (votes[c * NITEMS + item] > max_votes) {
          max_votes = votes[c * NITEMS + item];
          best_class = c;
        }
      out[row] = best_class;
    }
  }
};

template <int NITEMS, leaf_value_desc_t leaf_payload_type, typename output_type,
          typename storage_type>
__global__ void infer_k(storage_type forest, predict_params params) {
  // cache the row for all threads to reuse
  extern __shared__ char smem[];
  float* sdata = (float*)smem;
  size_t rid = blockIdx.x * NITEMS;
  for (int j = 0; j < NITEMS; ++j) {
    for (int i = threadIdx.x; i < params.num_cols; i += blockDim.x) {
      size_t row = rid + j;
      sdata[j * params.num_cols + i] =
        row < params.num_rows ? params.data[row * params.num_cols + i] : 0.0f;
    }
  }

  tree_aggregator_t<NITEMS, leaf_payload_type, output_type> acc(
    params.num_output_classes, sdata + params.num_cols * NITEMS);

  __syncthreads();  // for both row cache init and acc init

  // one block works on NITEMS rows and the whole forest
  for (int j = threadIdx.x; j < forest.num_trees(); j += blockDim.x) {
    acc.accumulate(
      infer_one_tree<NITEMS, val_t>(forest[j], sdata, params.num_cols));
  }
  // compute most probable class. in cuML RF, output is class label,
  // hence, no-predicted class edge case doesn't apply
  if (!params.predict_proba)
    acc.finalize_class_label(params.preds, params.num_rows);
  else
    acc.finalize(params.preds, params.num_rows);
}

template <leaf_value_desc_t leaf_payload_type, typename output_type,
          typename storage_type>
void infer_k_launcher(storage_type forest, predict_params params,
                      hipStream_t stream) {
  const int MAX_BATCH_ITEMS = 4;
  params.max_items =
    params.algo == algo_t::BATCH_TREE_REORG ? MAX_BATCH_ITEMS : 1;

  int shared_mem_per_item = sizeof(float) * params.num_cols;
  switch (leaf_payload_type) {
    case INT_CLASS_LABEL:
      // class vote histogram, while inferring trees
      shared_mem_per_item += sizeof(int) * params.num_output_classes;
      break;
    case FLOAT_SCALAR:
      // CUB workspace should fit itself, and we don't need
      // the row by the time CUB is used
      break;
  }
  int num_items = params.max_shm / shared_mem_per_item;
  if (num_items == 0) {
    int max_cols = params.max_shm / sizeof(float);
    ASSERT(false, "p.num_cols == %d: too many features, only %d allowed%s",
           params.num_cols, max_cols, leaf_payload_type == INT_CLASS_LABEL ?
           "(accounting for shared class vote histogram)" : "");
  }
  num_items = std::min(num_items, params.max_items);
  int num_blocks = ceildiv(int(params.num_rows), num_items);
  int shm_sz = num_items * shared_mem_per_item;
  switch (num_items) {
    case 1:
      infer_k<1, leaf_payload_type, output_type>
        <<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
      break;
    case 2:
      infer_k<2, leaf_payload_type, output_type>
        <<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
      break;
    case 3:
      infer_k<3, leaf_payload_type, output_type>
        <<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
      break;
    case 4:
      infer_k<4, leaf_payload_type, output_type>
        <<<num_blocks, FIL_TPB, shm_sz, stream>>>(forest, params);
      break;
    default:
      ASSERT(false, "internal error: nitems > 4");
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream) {
  printf("infer::num_output_classes = %u\n", params.num_output_classes);
  switch (params.leaf_payload_type) {
    case FLOAT_SCALAR:
      ASSERT(params.num_output_classes <= 2,
             "wrong leaf payload for multi-class (>2) inference");
      infer_k_launcher<FLOAT_SCALAR, float, storage_type>(forest, params,
                                                          stream);
      break;
    case INT_CLASS_LABEL:
      infer_k_launcher<INT_CLASS_LABEL, unsigned int, storage_type>(
        forest, params, stream);
      break;
    default:
      ASSERT(false, "unknown leaf_payload_type");
  }
}

template void infer<dense_storage>(dense_storage forest, predict_params params,
                                   hipStream_t stream);
template void infer<sparse_storage>(sparse_storage forest,
                                    predict_params params, hipStream_t stream);

}  // namespace fil
}  // namespace ML
#undef __forceinline__
