#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>

#include <cuml/fil/multi_sum.cuh>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;

// vec wraps float[N] for hipcub::BlockReduce
template <int N, typename T>
struct vec;

template <typename BinaryOp>
struct Vectorized {
  BinaryOp op;
  __device__ Vectorized(BinaryOp op_) : op(op_) {}
  template <int NITEMS, typename T>
  constexpr __host__ __device__ __forceinline__ vec<NITEMS, T> operator()(
    vec<NITEMS, T> a, vec<NITEMS, T> b) const {
    vec<NITEMS, T> c;
#pragma unroll
    for (int i = 0; i < NITEMS; i++) c[i] = op(a[i], b[i]);
    return c;
  }
};
template <typename BinaryOp>
constexpr __host__ __device__ Vectorized<BinaryOp> vectorized(BinaryOp op) {
  return op;
}

template <int N, typename T>
struct vec {
  static const int NITEMS = N;
  T data[N];
  explicit __host__ __device__ vec(T t) {
#pragma unroll
    for (int i = 0; i < N; ++i) data[i] = t;
  }
  __host__ __device__ vec() : vec(T()) {}
  __host__ __device__ T& operator[](int i) { return data[i]; }
  __host__ __device__ T operator[](int i) const { return data[i]; }
  friend __host__ __device__ vec<N, T> operator+(const vec<N, T>& a,
                                                 const vec<N, T>& b) {
    return vectorized(hipcub::Sum())(a, b);
  }
  friend __host__ __device__ void operator+=(vec<N, T>& a, const vec<N, T>& b) {
    a = a + b;
  }
  template <typename Vec>
  friend __host__ __device__ vec<N, T> operator/(vec<N, T>& a, const Vec& b) {
    return vectorized(thrust::divides<T>())(a, vec<N, T>(b));
  }
  template <typename Vec>
  friend __host__ __device__ void operator/=(vec<N, T>& a, const Vec& b) {
    a = a / b;
  }
};

struct best_margin_label : hipcub::KeyValuePair<int, float> {
  __host__ __device__ best_margin_label(hipcub::KeyValuePair<int, float> pair)
    : hipcub::KeyValuePair<int, float>(pair) {}
  __host__ __device__ best_margin_label(int c = 0, float f = -INFINITY)
    : hipcub::KeyValuePair<int, float>({c, f}) {}
};

template <int NITEMS>
__device__ __forceinline__ vec<NITEMS, best_margin_label> to_vec(
  int c, vec<NITEMS, float> margin) {
  vec<NITEMS, best_margin_label> ret;
#pragma unroll
  for (int i = 0; i < NITEMS; ++i) ret[i] = best_margin_label(c, margin[i]);
  return ret;
}

template <int NITEMS, typename output_type, typename tree_type>
__device__ __forceinline__ vec<NITEMS, output_type> infer_one_tree(
  tree_type tree, float* sdata, int cols) {
  int curr[NITEMS];
  int mask = (1 << NITEMS) - 1;  // all active
  for (int j = 0; j < NITEMS; ++j) curr[j] = 0;
  do {
#pragma unroll
    for (int j = 0; j < NITEMS; ++j) {
      auto n = tree[curr[j]];
      if (n.is_leaf()) {
        mask &= ~(1 << j);
        continue;
      }
      float val = sdata[j * cols + n.fid()];
      bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
      curr[j] = n.left(curr[j]) + cond;
    }
  } while (mask != 0);
  vec<NITEMS, output_type> out;
#pragma unroll
  for (int j = 0; j < NITEMS; ++j) {
    /** dependent names are not considered templates by default,
        unless it's a member of a current [template] instantiation.
        alternatively, could have used .base_node::output<... */
    out[j] = tree[curr[j]].template output<output_type>();
  }
  return out;
}

template <typename output_type, typename tree_type>
__device__ __forceinline__ vec<1, output_type> infer_one_tree(tree_type tree,
                                                              float* sdata,
                                                              int cols) {
  int curr = 0;
  for (;;) {
    auto n = tree[curr];
    if (n.is_leaf()) break;
    float val = sdata[n.fid()];
    bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
    curr = n.left(curr) + cond;
  }
  vec<1, output_type> out;
  out[0] = tree[curr].base_node::output<output_type>();
  return out;
}

/**
The shared memory requirements for finalization stage may differ based
on the set of PTX architectures the kernels were compiled for, as well as 
the CUDA compute capability of the device chosen for computation.

TODO (levsnv): run a test kernel during forest init to determine the compute capability
chosen for the inference, for an accurate sizeof(BlockReduce::TempStorage),
which is used in determining max NITEMS or max input data columns.

600 is the __CUDA_ARCH__ for Pascal (6.0) GPUs, which is not defined in
host code.
6.0 is the earliest compute capability supported by FIL and RAPIDS in general.
See https://rapids.ai/start.html as well as cmake defaults.
*/
// values below are defaults as of this change.
template <int NITEMS>
size_t block_reduce_footprint_host() {
  return sizeof(typename hipcub::BlockReduce<vec<NITEMS, float>, FIL_TPB,
                                          hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1,
                                          1, 600>::TempStorage);
}

template <int NITEMS>
size_t block_reduce_best_class_footprint_host() {
  return sizeof(
    typename hipcub::BlockReduce<vec<NITEMS, best_margin_label>, FIL_TPB,
                              hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 1, 1,
                              600>::TempStorage);
}

// the device template should achieve the best performance, using up-to-date
// CUB defaults
template <typename T, typename BinaryOp>
__device__ __forceinline__ T block_reduce(T value, BinaryOp op, void* storage) {
  typedef hipcub::BlockReduce<T, FIL_TPB> BlockReduceT;
  return BlockReduceT(*(typename BlockReduceT::TempStorage*)storage)
    .Reduce(value, op, blockDim.x);
}

template <int NITEMS,
          leaf_algo_t leaf_algo>  // = FLOAT_UNARY_BINARY
struct tree_aggregator_t {
  vec<NITEMS, float> acc;
  void* tmp_storage;

  /** shared memory footprint of the accumulator during
  the finalization of forest inference kernel, when infer_k output
  value is computed.
  num_classes is used for other template parameters */
  static size_t smem_finalize_footprint(size_t data_row_size, int num_classes,
                                        bool predict_proba) {
    return block_reduce_footprint_host<NITEMS>();
  }

  /** shared memory footprint of the accumulator during
  the accumulation of forest inference, when individual trees
  are inferred and partial aggregates are accumulated.
  num_classes is used for other template parameters */
  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  /** 
  num_classes is used for other template parameters */
  __device__ __forceinline__ tree_aggregator_t(int num_classes,
                                               void* shared_workspace, size_t)
    : tmp_storage(shared_workspace) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int output_stride,
                                           output_t transform, int num_trees) {
    __syncthreads();
    acc = block_reduce(acc, vectorized(hipcub::Sum()), tmp_storage);
    if (threadIdx.x > 0) return;
#pragma unroll
    for (int row = 0; row < NITEMS; ++row)
      if (row < num_rows) out[row * output_stride] = acc[row];
  }
};

// tmp_storage may overlap shared memory addressed by [begin, end)
// allreduce_shmem ensures no race conditions
template <typename Iterator, typename BinaryOp>
__device__ __forceinline__ auto allreduce_shmem(Iterator begin, Iterator end,
                                                BinaryOp op,
                                                void* tmp_storage) {
  typedef typename std::iterator_traits<Iterator>::value_type value_type;
  value_type thread_partial;
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    thread_partial = op(thread_partial, *it);
  __syncthreads();  // free shared memory [begin, end)
  auto res = block_reduce(thread_partial, op, tmp_storage);
  // broadcast sum to all threads
  __syncthreads();  // free up tmp_storage
  if (threadIdx.x == 0) *(value_type*)tmp_storage = res;
  __syncthreads();
  return *(value_type*)tmp_storage;
}

// *begin and *end shall be struct vec
// tmp_storage may overlap shared memory addressed by [begin, end)
template <typename Iterator>
__device__ __forceinline__ void write_best_class(Iterator begin, Iterator end,
                                                 void* tmp_storage, float* out,
                                                 int num_rows) {
  // reduce per-class candidate margins to one best class candidate
  // per thread (for each of the NITEMS rows)
  auto best = vec<begin->NITEMS, best_margin_label>();
  for (int c = threadIdx.x; c < end - begin; c += blockDim.x)
    best = vectorized(hipcub::ArgMax())(best, to_vec(c, begin[c]));
  // [begin, end) may overlap tmp_storage
  __syncthreads();
  // find best class per block (for each of the NITEMS rows)
  best = block_reduce(best, vectorized(hipcub::ArgMax()), tmp_storage);
  // write it out to global memory
  if (threadIdx.x > 0) return;
#pragma unroll
  for (int row = 0; row < best.NITEMS; ++row)
    if (row < num_rows) out[row] = best[row].key;
}

/// needed for softmax
__device__ float shifted_exp(float margin, float max) {
  return expf(margin - max);
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
template <typename Iterator>
__device__ __forceinline__ void block_softmax(Iterator begin, Iterator end,
                                              void* tmp_storage) {
  // subtract max before exponentiating for numerical stability
  typedef typename std::iterator_traits<Iterator>::value_type value_type;
  value_type max =
    allreduce_shmem(begin, end, vectorized(hipcub::Max()), tmp_storage);
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    *it = vectorized(shifted_exp)(*it, max);
  // sum of exponents
  value_type soe =
    allreduce_shmem(begin, end, vectorized(hipcub::Sum()), tmp_storage);
  // softmax phase 2: normalization
  for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
    *it /= soe;
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
template <typename Iterator>
__device__ __forceinline__ void normalize_softmax_and_write(
  Iterator begin, Iterator end, output_t transform, int trees_per_class,
  void* tmp_storage, float* out, int num_rows) {
  if ((transform & output_t::AVG) != 0) {
    for (Iterator it = begin + threadIdx.x; it < end; it += blockDim.x)
      *it /= trees_per_class;
  }
  if ((transform & output_t::SOFTMAX) != 0)
    block_softmax(begin, end, tmp_storage);
// write result to global memory
#pragma unroll
  for (int row = 0; row < begin->NITEMS; ++row) {
    for (int c = threadIdx.x; c < end - begin; c += blockDim.x)
      if (row < num_rows) out[row * (end - begin) + c] = begin[c][row];
  }
}

// *begin and *end shall be struct vec
// tmp_storage may NOT overlap shared memory addressed by [begin, end)
// in case num_outputs > 1
template <typename Iterator>
__device__ __forceinline__ void class_margins_to_global_memory(
  Iterator begin, Iterator end, output_t transform, int trees_per_class,
  void* tmp_storage, float* out, int num_rows, int num_outputs) {
  if (num_outputs == 1) {  // will output class
    // reduce per-class candidate margins to one best class candidate
    // per thread (for each of the NITEMS rows)
    write_best_class(begin, end, tmp_storage, out, num_rows);
  } else {  // output softmax-ed margin
    normalize_softmax_and_write(begin, end, transform, trees_per_class,
                                tmp_storage, out, num_rows);
  }
}

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_FEW_CLASSES> {
  vec<NITEMS, float> acc;
  void* tmp_storage;
  int num_classes;

  static size_t smem_finalize_footprint(size_t data_row_size, int num_classes,
                                        bool predict_proba) {
    size_t phase1 =
      (FIL_TPB - FIL_TPB % num_classes) * sizeof(vec<NITEMS, float>);
    size_t phase2 = predict_proba
                      ? block_reduce_footprint_host<NITEMS>()
                      : block_reduce_best_class_footprint_host<NITEMS>();
    return predict_proba ? phase1 + phase2 : std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) { return 0; }

  __device__ __forceinline__ tree_aggregator_t(int num_classes_,
                                               void* shared_workspace, size_t)
    : tmp_storage(shared_workspace), num_classes(num_classes_) {}

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    acc += single_tree_prediction;
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs, output_t transform,
                                           int num_trees) {
    __syncthreads();  // free up input row
    // load margin into shared memory
    auto per_thread = (vec<NITEMS, float>*)tmp_storage;
    per_thread[threadIdx.x] = acc;
    __syncthreads();
    acc = multi_sum<6>(per_thread, num_classes, blockDim.x / num_classes);
    if (threadIdx.x < num_classes) per_thread[threadIdx.x] = acc;
    __syncthreads();  // per_thread needs to be fully populated

    void* storage = num_outputs > 1 ? per_thread + num_classes : tmp_storage;
    class_margins_to_global_memory(per_thread, per_thread + num_classes,
                                   transform, num_trees / num_classes, storage,
                                   out, num_rows, num_outputs);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, GROVE_PER_CLASS_MANY_CLASSES> {
  vec<NITEMS, float> acc;
  /// at first, per class margin, then, possibly, different softmax partials
  vec<NITEMS, float>* per_class_value;
  void* tmp_storage;
  int num_classes;

  static size_t smem_finalize_footprint(size_t data_row_size, int num_classes,
                                        bool predict_proba) {
    size_t phase1 = data_row_size + smem_accumulate_footprint(num_classes);
    size_t phase2 = predict_proba
                      ? block_reduce_footprint_host<NITEMS>()
                      : block_reduce_best_class_footprint_host<NITEMS>();
    return predict_proba ? phase1 + phase2 : std::max(phase1, phase2);
  }

  static size_t smem_accumulate_footprint(int num_classes) {
    return num_classes * sizeof(vec<NITEMS, float>);
  }

  __device__ __forceinline__ tree_aggregator_t(int num_classes_,
                                               void* shared_workspace,
                                               size_t data_row_size)
    : per_class_value(
        (vec<NITEMS, float>*)((char*)shared_workspace + data_row_size)),
      tmp_storage(shared_workspace),
      num_classes(num_classes_) {
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x)
      per_class_value[c] = vec<NITEMS, float>(0);
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void accumulate(
    vec<NITEMS, float> single_tree_prediction, int tree) {
    // since threads are assigned to consecutive classes, no need for atomics
    per_class_value[tree % num_classes] += single_tree_prediction;
    // __syncthreads() is called in infer_k
  }

  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs, output_t transform,
                                           int num_trees) {
    void* storage =
      num_outputs > 1 ? per_class_value + num_classes : tmp_storage;
    class_margins_to_global_memory(
      per_class_value, per_class_value + num_classes, transform,
      num_trees / num_classes, storage, out, num_rows, num_outputs);
  }
};

template <int NITEMS>
struct tree_aggregator_t<NITEMS, CATEGORICAL_LEAF> {
  // could switch to unsigned short to save shared memory
  // provided raft::myAtomicAdd(short*) simulated with appropriate shifts
  int* votes;
  int num_classes;

  static size_t smem_finalize_footprint(int data_row_size, int num_classes,
                                        bool predict_proba) {
    return data_row_size + sizeof(int) * num_classes * NITEMS;
  }
  static size_t smem_accumulate_footprint(int num_classes) {
    return sizeof(int) * num_classes * NITEMS;
  }

  __device__ __forceinline__ tree_aggregator_t(int num_classes_,
                                               void* shared_workspace,
                                               size_t data_row_size)
    : num_classes(num_classes_),
      votes((int*)(data_row_size + (char*)shared_workspace)) {
    for (int c = threadIdx.x; c < num_classes; c += FIL_TPB * NITEMS)
#pragma unroll
      for (int item = 0; item < NITEMS; ++item) votes[c * NITEMS + item] = 0;
    // __syncthreads() is called in infer_k
  }
  __device__ __forceinline__ void accumulate(
    vec<NITEMS, int> single_tree_prediction, int tree) {
#pragma unroll
    for (int item = 0; item < NITEMS; ++item)
      raft::myAtomicAdd(votes + single_tree_prediction[item] * NITEMS + item,
                        1);
  }
  // class probabilities or regression. for regression, num_classes
  // is just the number of outputs for each data instance
  __device__ __forceinline__ void finalize_multiple_outputs(float* out,
                                                            int num_rows) {
    __syncthreads();
    for (int c = threadIdx.x; c < num_classes; c += blockDim.x) {
#pragma unroll
      for (int row = 0; row < num_rows; ++row)
        out[row * num_classes + c] = votes[c * NITEMS + row];
    }
  }
  // using this when predicting a single class label, as opposed to sparse class vector
  // or class probabilities or regression
  __device__ __forceinline__ void finalize_class_label(float* out,
                                                       int num_rows) {
    __syncthreads();
    int item = threadIdx.x;
    int row = item;
    if (item < NITEMS && row < num_rows) {
      int max_votes = 0;
      int best_class = 0;
      for (int c = 0; c < num_classes; ++c) {
        if (votes[c * NITEMS + item] > max_votes) {
          max_votes = votes[c * NITEMS + item];
          best_class = c;
        }
      }
      out[row] = best_class;
    }
  }
  __device__ __forceinline__ void finalize(float* out, int num_rows,
                                           int num_outputs, output_t transform,
                                           int num_trees) {
    if (num_outputs > 1) {
      // only supporting num_outputs == num_classes
      finalize_multiple_outputs(out, num_rows);
    } else {
      finalize_class_label(out, num_rows);
    }
  }
};

template <int NITEMS, leaf_algo_t leaf_algo, class storage_type>
__global__ void infer_k(storage_type forest, predict_params params) {
  extern __shared__ char smem[];
  float* sdata = (float*)smem;
  for (size_t block_row0 = blockIdx.x * NITEMS; block_row0 < params.num_rows;
       block_row0 += NITEMS * gridDim.x) {
    // cache the row for all threads to reuse
    for (size_t j = 0; j < NITEMS; ++j) {
      size_t row = block_row0 + j;
#pragma unroll
      for (int col = threadIdx.x; col < params.num_cols; col += blockDim.x) {
        sdata[j * params.num_cols + col] =
          row < params.num_rows ? params.data[row * params.num_cols + col]
                                : 0.0f;
      }
    }

    tree_aggregator_t<NITEMS, leaf_algo> acc(
      params.num_classes, sdata, params.num_cols * NITEMS * sizeof(float));

    __syncthreads();  // for both row cache init and acc init

    // one block works on NITEMS rows and the whole forest
    for (int j = threadIdx.x; j - threadIdx.x < forest.num_trees();
         j += blockDim.x) {
      /* j - threadIdx.x < forest.num_trees() is a necessary but block-uniform
         condition for "j < forest.num_trees()". It lets use __syncthreads()
         and is made exact below.
      */
      if (j < forest.num_trees()) {
        acc.accumulate(infer_one_tree<NITEMS, leaf_output_t<leaf_algo>::T>(
                         forest[j], sdata, params.num_cols),
                       j);
      }
      if (leaf_algo == GROVE_PER_CLASS_MANY_CLASSES) __syncthreads();
    }
    acc.finalize(params.preds + params.num_outputs * block_row0,
                 min((size_t)NITEMS, params.num_rows - block_row0),
                 params.num_outputs, params.transform, forest.num_trees());
    __syncthreads();  // free up acc's shared memory resources for next row set
  }
}

template <int NITEMS, leaf_algo_t leaf_algo>
size_t get_smem_footprint(predict_params params) {
  size_t finalize_footprint =
    tree_aggregator_t<NITEMS, leaf_algo>::smem_finalize_footprint(
      sizeof(float) * params.num_cols * NITEMS, params.num_classes,
      params.num_outputs == params.num_classes);
  size_t accumulate_footprint =
    sizeof(float) * params.num_cols * NITEMS +
    tree_aggregator_t<NITEMS, leaf_algo>::smem_accumulate_footprint(
      params.num_classes);

  return std::max(accumulate_footprint, finalize_footprint);
}

template <leaf_algo_t leaf_algo, typename storage_type>
void infer_k_launcher(storage_type forest, predict_params params,
                      hipStream_t stream, int blockdim_x) {
  const int MAX_BATCH_ITEMS = 4;
  params.max_items =
    params.algo == algo_t::BATCH_TREE_REORG ? MAX_BATCH_ITEMS : 1;

  /** searching for the most items per block while respecting the shared
  * memory limits creates a full linear programming problem.
  * solving it in a single equation looks less tractable than this */
  int num_items = 0;
  size_t shm_sz = 0;
  for (int nitems = 1; nitems <= params.max_items; ++nitems) {
    size_t peak_footprint;
    switch (nitems) {
      case 1:
        peak_footprint = get_smem_footprint<1, leaf_algo>(params);
        break;
      case 2:
        peak_footprint = get_smem_footprint<2, leaf_algo>(params);
        break;
      case 3:
        peak_footprint = get_smem_footprint<3, leaf_algo>(params);
        break;
      case 4:
        peak_footprint = get_smem_footprint<4, leaf_algo>(params);
        break;
      default:
        ASSERT(false, "internal error: nitems > 4");
    }
    // for data row
    if (peak_footprint <= params.max_shm) {
      num_items = nitems;
      shm_sz = peak_footprint;
    }
  }
  if (num_items == 0) {
    int given_num_cols = params.num_cols;
    // starting with maximum that might fit in shared memory, in case
    // given_num_cols is a random large int
    params.num_cols = params.max_shm / sizeof(float);
    // since we're crashing, this will not take too long
    while (params.num_cols > 0 &&
           get_smem_footprint<1, leaf_algo>(params) > params.max_shm) {
      --params.num_cols;
    }
    ASSERT(false, "p.num_cols == %d: too many features, only %d allowed",
           given_num_cols, params.num_cols);
  }
  params.num_blocks = params.num_blocks != 0
                        ? params.num_blocks
                        : raft::ceildiv(int(params.num_rows), num_items);
  switch (num_items) {
    case 1:
      infer_k<1, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 2:
      infer_k<2, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 3:
      infer_k<3, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    case 4:
      infer_k<4, leaf_algo>
        <<<params.num_blocks, blockdim_x, shm_sz, stream>>>(forest, params);
      break;
    default:
      ASSERT(false, "internal error: nitems > 4");
  }
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename storage_type>
void infer(storage_type forest, predict_params params, hipStream_t stream) {
  switch (params.leaf_algo) {
    case FLOAT_UNARY_BINARY:
      infer_k_launcher<FLOAT_UNARY_BINARY>(forest, params, stream, FIL_TPB);
      break;
    case GROVE_PER_CLASS:
      if (params.num_classes > FIL_TPB) {
        params.leaf_algo = GROVE_PER_CLASS_MANY_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_MANY_CLASSES>(forest, params, stream,
                                                       FIL_TPB);
      } else {
        params.leaf_algo = GROVE_PER_CLASS_FEW_CLASSES;
        infer_k_launcher<GROVE_PER_CLASS_FEW_CLASSES>(
          forest, params, stream, FIL_TPB - FIL_TPB % params.num_classes);
      }
      break;
    case CATEGORICAL_LEAF:
      infer_k_launcher<CATEGORICAL_LEAF>(forest, params, stream, FIL_TPB);
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_algo");
  }
}

template void infer<dense_storage>(dense_storage forest, predict_params params,
                                   hipStream_t stream);
template void infer<sparse_storage16>(sparse_storage16 forest,
                                      predict_params params,
                                      hipStream_t stream);
template void infer<sparse_storage8>(sparse_storage8 forest,
                                     predict_params params,
                                     hipStream_t stream);

}  // namespace fil
}  // namespace ML
