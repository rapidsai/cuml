#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file fil.cu implements forest inference */

#include <omp.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <treelite/c_api.h>
#include <treelite/tree.h>
#include <algorithm>
#include <cmath>
#include <limits>
#include <stack>
#include <utility>

#include <cuml/fil/fil.h>
#include <raft/cudart_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;
namespace tl = treelite;

__host__ __device__ float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

/** performs additional transformations on the array of forest predictions
    (preds) of size n; the transformations are defined by output, and include
    averaging (multiplying by inv_num_trees), adding global_bias (always done),
    sigmoid and applying threshold. in case of complement_proba,
    fills in the complement probability */
__global__ void transform_k(float* preds, size_t n, output_t output,
                            float inv_num_trees, float threshold,
                            float global_bias, bool complement_proba) {
  size_t i = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (i >= n) return;
  if (complement_proba && i % 2 != 0) return;

  float result = preds[i];
  if ((output & output_t::AVG) != 0) result *= inv_num_trees;
  result += global_bias;
  if ((output & output_t::SIGMOID) != 0) result = sigmoid(result);
  // will not be done on CATEGORICAL_LEAF because the whole kernel will not run
  if ((output & output_t::CLASS) != 0) {
    result = result > threshold ? 1.0f : 0.0f;
  }
  // sklearn outputs numpy array in 'C' order, with the number of classes being last dimension
  // that is also the default order, so we should use the same one
  if (complement_proba) {
    preds[i] = 1.0f - result;
    preds[i + 1] = result;
  } else
    preds[i] = result;
}

struct forest {
  void init_n_items(int device) {
    int max_shm_std = 48 * 1024;  // 48 KiB
    /// the most shared memory a kernel can request on the GPU in question
    int max_shm = 0;
    CUDA_CHECK(hipDeviceGetAttribute(
      &max_shm, hipDeviceAttributeSharedMemPerBlockOptin, device));
    // TODO(canonizer): use >48KiB shared memory if available
    max_shm = std::min(max_shm, max_shm_std);

    // searching for the most items per block while respecting the shared
    // memory limits creates a full linear programming problem.
    // solving it in a single equation looks less tractable than this
    for (bool predict_proba : {false, true}) {
      shmem_size_params& ssp_ = predict_proba ? proba_ssp_ : class_ssp_;
      ssp_.predict_proba = predict_proba;
      shmem_size_params ssp = ssp_;
      for (bool cols_in_shmem : {false, true}) {
        ssp.cols_in_shmem = cols_in_shmem;
        for (ssp.n_items = 1;
             ssp.n_items <= (algo_ == algo_t::BATCH_TREE_REORG ? 4 : 1);
             ++ssp.n_items) {
          ssp.compute_smem_footprint();
          if (ssp.shm_sz < max_shm) ssp_ = ssp;
        }
      }
      ASSERT(max_shm >= ssp_.shm_sz,
             "FIL out of shared memory. Perhaps the maximum number of \n"
             "supported classes is exceeded? 5'000 would still be safe.");
    }
  }

  void init_fixed_block_count(int device, int blocks_per_sm) {
    int max_threads_per_sm, sm_count;
    CUDA_CHECK(hipDeviceGetAttribute(
      &max_threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, device));
    int max_blocks_per_sm = max_threads_per_sm / FIL_TPB;
    ASSERT(blocks_per_sm <= max_blocks_per_sm,
           "on this GPU, FIL blocks_per_sm cannot exceed %d",
           max_blocks_per_sm);
    CUDA_CHECK(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount,
                                      device));
    fixed_block_count_ = blocks_per_sm * sm_count;
  }

  void init_common(const raft::handle_t& h, const forest_params_t* params) {
    depth_ = params->depth;
    num_trees_ = params->num_trees;
    algo_ = params->algo;
    output_ = params->output;
    threshold_ = params->threshold;
    global_bias_ = params->global_bias;
    proba_ssp_.leaf_algo = params->leaf_algo;
    proba_ssp_.num_cols = params->num_cols;
    proba_ssp_.num_classes = params->num_classes;
    class_ssp_ = proba_ssp_;

    int device = h.get_device();
    init_n_items(device);  // n_items takes priority over blocks_per_sm
    init_fixed_block_count(device, params->blocks_per_sm);
  }

  virtual void infer(predict_params params, hipStream_t stream) = 0;

  void predict(const raft::handle_t& h, float* preds, const float* data,
               size_t num_rows, bool predict_proba) {
    // Initialize prediction parameters.
    predict_params params(predict_proba ? proba_ssp_ : class_ssp_);
    params.algo = algo_;
    params.preds = preds;
    params.data = data;
    params.num_rows = num_rows;
    // ignored unless predict_proba is true and algo is GROVE_PER_CLASS
    params.transform = output_;
    // fixed_block_count_ == 0 means the number of thread blocks is
    // proportional to the number of rows
    params.num_blocks = fixed_block_count_;

    /**
    The binary classification / regression (FLOAT_UNARY_BINARY) predict_proba() works as follows
      (always 2 outputs):
    RAW: output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error
    write the output of the previous stages and its complement

    The binary classification / regression (FLOAT_UNARY_BINARY) predict() works as follows
      (always 1 output):
    RAW (no values set): output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: apply threshold (equivalent to choosing best class)
    SOFTMAX is set: error
    
    The multi-class classification / regression (CATEGORICAL_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error
    
    The multi-class classification / regression (CATEGORICAL_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest probability, else output label 0.
    SOFTMAX is set: error
    All other flags (AVG, SIGMOID, CLASS) are ignored
    
    The multi-class classification / regression (GROVE_PER_CLASS) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid; if SOFTMAX is also set: error
    CLASS is set: ignored
    SOFTMAX is set: softmax is applied after averaging and global_bias

    The multi-class classification / regression (GROVE_PER_CLASS) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS, SOFTMAX) are ignored
    */
    output_t ot = output_;
    // Treelite applies bias before softmax, but we do after.
    // Simulating treelite order, which cancels out bias.
    // If non-proba prediction used, it still will not matter
    // for the same reason softmax will not.
    float global_bias = (ot & output_t::SOFTMAX) != 0 ? 0.0f : global_bias_;
    bool complement_proba = false, do_transform;

    if (predict_proba) {
      // no threshold on probabilities
      ot = output_t(ot & ~output_t::CLASS);

      switch (params.leaf_algo) {
        case leaf_algo_t::FLOAT_UNARY_BINARY:
          params.num_outputs = 2;
          complement_proba = true;
          do_transform = true;
          break;
        case leaf_algo_t::GROVE_PER_CLASS:
          // for GROVE_PER_CLASS, averaging happens in infer_k
          ot = output_t(ot & ~output_t::AVG);
          params.num_outputs = params.num_classes;
          do_transform = ot != output_t::RAW && ot != output_t::SOFTMAX ||
                         global_bias != 0.0f;
          break;
        case leaf_algo_t::CATEGORICAL_LEAF:
          params.num_outputs = params.num_classes;
          do_transform = ot != output_t::RAW || global_bias_ != 0.0f;
          break;
        default:
          ASSERT(false, "internal error: invalid leaf_algo_");
      }
    } else {
      if (params.leaf_algo == leaf_algo_t::FLOAT_UNARY_BINARY) {
        do_transform = ot != output_t::RAW || global_bias_ != 0.0f;
      } else {
        // GROVE_PER_CLASS, CATEGORICAL_LEAF: moot since choosing best class and
        // all transforms are monotonic. also, would break current code
        do_transform = false;
      }
      params.num_outputs = 1;
    }

    // Predict using the forest.
    hipStream_t stream = h.get_stream();
    infer(params, stream);

    if (do_transform) {
      size_t num_values_to_transform =
        (size_t)num_rows * (size_t)params.num_outputs;
      transform_k<<<raft::ceildiv(num_values_to_transform, (size_t)FIL_TPB),
                    FIL_TPB, 0, stream>>>(
        preds, num_values_to_transform, ot,
        num_trees_ > 0 ? (1.0f / num_trees_) : 1.0f, threshold_, global_bias,
        complement_proba);
      CUDA_CHECK(hipPeekAtLastError());
    }
  }

  virtual void free(const raft::handle_t& h) = 0;
  virtual ~forest() {}

  int num_trees_ = 0;
  int depth_ = 0;
  algo_t algo_ = algo_t::NAIVE;
  output_t output_ = output_t::RAW;
  float threshold_ = 0.5;
  float global_bias_ = 0;
  shmem_size_params class_ssp_, proba_ssp_;
  int fixed_block_count_ = 0;
};

struct dense_forest : forest {
  void transform_trees(const dense_node* nodes) {
    /* Populate node information:
       For each tree, the nodes are still stored in the breadth-first,
       left-to-right order. However, instead of storing the nodes of the same
       tree adjacently, it uses a different layout. In this layout, the roots
       of all trees (node 0) are stored first, followed by left children of
       the roots of all trees (node 1), followed by the right children of the
       roots of all trees (node 2), and so on.
    */
    int global_node = 0;
    for (int tree = 0; tree < num_trees_; ++tree) {
      int tree_node = 0;
      // the counters `level` and `branch` are not used for computing node
      // indices, they are only here to highlight the node ordering within
      // each tree
      for (int level = 0; level <= depth_; ++level) {
        for (int branch = 0; branch < 1 << level; ++branch) {
          h_nodes_[tree_node * num_trees_ + tree] = nodes[global_node];
          ++tree_node;
          ++global_node;
        }
      }
    }
  }

  void init(const raft::handle_t& h, const dense_node* nodes,
            const forest_params_t* params) {
    init_common(h, params);
    if (algo_ == algo_t::NAIVE) algo_ = algo_t::BATCH_TREE_REORG;

    int num_nodes = forest_num_nodes(num_trees_, depth_);
    nodes_ = (dense_node*)h.get_device_allocator()->allocate(
      sizeof(dense_node) * num_nodes, h.get_stream());
    h_nodes_.resize(num_nodes);
    if (algo_ == algo_t::NAIVE) {
      std::copy(nodes, nodes + num_nodes, h_nodes_.begin());
    } else {
      transform_trees(nodes);
    }
    CUDA_CHECK(hipMemcpyAsync(nodes_, h_nodes_.data(),
                               num_nodes * sizeof(dense_node),
                               hipMemcpyHostToDevice, h.get_stream()));
    // copy must be finished before freeing the host data
    CUDA_CHECK(hipStreamSynchronize(h.get_stream()));
    h_nodes_.clear();
    h_nodes_.shrink_to_fit();
  }

  virtual void infer(predict_params params, hipStream_t stream) override {
    dense_storage forest(nodes_, num_trees_,
                         algo_ == algo_t::NAIVE ? tree_num_nodes(depth_) : 1,
                         algo_ == algo_t::NAIVE ? 1 : num_trees_);
    fil::infer(forest, params, stream);
  }

  virtual void free(const raft::handle_t& h) override {
    int num_nodes = forest_num_nodes(num_trees_, depth_);
    h.get_device_allocator()->deallocate(nodes_, sizeof(dense_node) * num_nodes,
                                         h.get_stream());
  }

  dense_node* nodes_ = nullptr;
  thrust::host_vector<dense_node> h_nodes_;
};

template <typename node_t>
struct sparse_forest : forest {
  void init(const raft::handle_t& h, const int* trees, const node_t* nodes,
            const forest_params_t* params) {
    init_common(h, params);
    if (algo_ == algo_t::ALGO_AUTO) algo_ = algo_t::NAIVE;
    depth_ = 0;  // a placeholder value
    num_nodes_ = params->num_nodes;

    // trees
    trees_ = (int*)h.get_device_allocator()->allocate(sizeof(int) * num_trees_,
                                                      h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(trees_, trees, sizeof(int) * num_trees_,
                               hipMemcpyHostToDevice, h.get_stream()));

    // nodes
    nodes_ = (node_t*)h.get_device_allocator()->allocate(
      sizeof(node_t) * num_nodes_, h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(nodes_, nodes, sizeof(node_t) * num_nodes_,
                               hipMemcpyHostToDevice, h.get_stream()));
  }

  virtual void infer(predict_params params, hipStream_t stream) override {
    sparse_storage<node_t> forest(trees_, nodes_, num_trees_);
    fil::infer(forest, params, stream);
  }

  void free(const raft::handle_t& h) override {
    h.get_device_allocator()->deallocate(trees_, sizeof(int) * num_trees_,
                                         h.get_stream());
    h.get_device_allocator()->deallocate(nodes_, sizeof(node_t) * num_nodes_,
                                         h.get_stream());
  }

  int num_nodes_ = 0;
  int* trees_ = nullptr;
  node_t* nodes_ = nullptr;
};

void check_params(const forest_params_t* params, bool dense) {
  if (dense) {
    ASSERT(params->depth >= 0, "depth must be non-negative for dense forests");
  } else {
    ASSERT(params->num_nodes >= 0,
           "num_nodes must be non-negative for sparse forests");
    ASSERT(params->algo == algo_t::NAIVE || params->algo == algo_t::ALGO_AUTO,
           "only ALGO_AUTO and NAIVE algorithms are supported "
           "for sparse forests");
  }
  ASSERT(params->num_trees >= 0, "num_trees must be non-negative");
  ASSERT(params->num_cols >= 0, "num_cols must be non-negative");
  switch (params->algo) {
    case algo_t::ALGO_AUTO:
    case algo_t::NAIVE:
    case algo_t::TREE_REORG:
    case algo_t::BATCH_TREE_REORG:
      break;
    default:
      ASSERT(false,
             "algo should be ALGO_AUTO, NAIVE, TREE_REORG or BATCH_TREE_REORG");
  }
  switch (params->leaf_algo) {
    case leaf_algo_t::FLOAT_UNARY_BINARY:
      if ((params->output & output_t::CLASS) != 0) {
        ASSERT(params->num_classes == 2,
               "only supporting binary"
               " classification using FLOAT_UNARY_BINARY");
      } else {
        ASSERT(params->num_classes == 1,
               "num_classes must be 1 for "
               "regression");
      }
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax does not make sense for leaf_algo == FLOAT_UNARY_BINARY");
      break;
    case leaf_algo_t::GROVE_PER_CLASS:
      ASSERT(params->num_classes > 2,
             "num_classes > 2 is required for leaf_algo == GROVE_PER_CLASS");
      ASSERT(params->num_trees % params->num_classes == 0,
             "num_classes must divide num_trees evenly for GROVE_PER_CLASS");
      break;
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == CATEGORICAL_LEAF");
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax not supported for leaf_algo == CATEGORICAL_LEAF");
      break;
    default:
      ASSERT(false,
             "leaf_algo must be FLOAT_UNARY_BINARY, CATEGORICAL_LEAF"
             " or GROVE_PER_CLASS");
  }
  if ((params->output & ~output_t::ALL_SET) != 0) {
    ASSERT(
      false,
      "output should be a combination of RAW, AVG, SIGMOID, CLASS and SOFTMAX");
  }
  ASSERT(~params->output & (output_t::SIGMOID | output_t::SOFTMAX),
         "combining softmax and sigmoid is not supported");
  ASSERT(params->blocks_per_sm >= 0, "blocks_per_sm must be nonnegative");
}

template <typename T, typename L>
int tree_root(const tl::Tree<T, L>& tree) {
  return 0;  // Treelite format assumes that the root is 0
}

template <typename T, typename L>
inline int max_depth(const tl::Tree<T, L>& tree) {
  // trees of this depth aren't used, so it most likely means bad input data,
  // e.g. cycles in the forest
  const int DEPTH_LIMIT = 500;
  int root_index = tree_root(tree);
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  stack.push(pair_t(root_index, 0));
  int max_depth = 0;
  while (!stack.empty()) {
    const pair_t& pair = stack.top();
    int node_id = pair.first;
    int depth = pair.second;
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      stack.push(pair_t(tree.LeftChild(node_id), depth + 1));
      node_id = tree.RightChild(node_id);
      depth++;
      ASSERT(depth < DEPTH_LIMIT,
             "depth limit reached, might be a cycle in the tree");
    }
    // only need to update depth for leaves
    max_depth = std::max(max_depth, depth);
  }
  return max_depth;
}

template <typename T, typename L>
int max_depth(const tl::ModelImpl<T, L>& model) {
  int depth = 0;
  const auto& trees = model.trees;
#pragma omp parallel for reduction(max : depth)
  for (size_t i = 0; i < trees.size(); ++i) {
    const auto& tree = trees[i];
    depth = std::max(depth, max_depth(tree));
  }
  return depth;
}

inline void adjust_threshold(float* pthreshold, int* tl_left, int* tl_right,
                             bool* default_left, tl::Operator comparison_op) {
  // in treelite (take left node if val [op] threshold),
  // the meaning of the condition is reversed compared to FIL;
  // thus, "<" in treelite corresonds to comparison ">=" used by FIL
  // https://github.com/dmlc/treelite/blob/master/include/treelite/tree.h#L243
  switch (comparison_op) {
    case tl::Operator::kLT:
      break;
    case tl::Operator::kLE:
      // x <= y is equivalent to x < y', where y' is the next representable float
      *pthreshold =
        std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
      break;
    case tl::Operator::kGT:
      // x > y is equivalent to x >= y', where y' is the next representable float
      // left and right still need to be swapped
      *pthreshold =
        std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
    case tl::Operator::kGE:
      // swap left and right
      std::swap(*tl_left, *tl_right);
      *default_left = !*default_left;
      break;
    default:
      ASSERT(false, "only <, >, <= and >= comparisons are supported");
  }
}

/** if the vector consists of zeros and a single one, return the position
for the one (assumed class label). Else, asserts false.
If the vector contains a NAN, asserts false */
template <typename L>
int find_class_label_from_one_hot(L* vector, int len) {
  bool found_label = false;
  int out;
  for (int i = 0; i < len; ++i) {
    if (vector[i] == static_cast<L>(1.0)) {
      ASSERT(!found_label, "label vector contains multiple 1.0f");
      out = i;
      found_label = true;
    } else {
      ASSERT(vector[i] == static_cast<L>(0.0),
             "label vector contains values other than 0.0 and 1.0");
    }
  }
  ASSERT(found_label, "did not find 1.0f in vector");
  return out;
}

template <typename fil_node_t, typename T, typename L>
void tl2fil_leaf_payload(fil_node_t* fil_node, const tl::Tree<T, L>& tl_tree,
                         int tl_node_id, const forest_params_t& forest_params) {
  auto vec = tl_tree.LeafVector(tl_node_id);
  switch (forest_params.leaf_algo) {
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(vec.size() == forest_params.num_classes,
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = find_class_label_from_one_hot(&vec[0], vec.size());
      break;
    case leaf_algo_t::FLOAT_UNARY_BINARY:
    case leaf_algo_t::GROVE_PER_CLASS:
      fil_node->val.f = static_cast<float>(tl_tree.LeafValue(tl_node_id));
      ASSERT(!tl_tree.HasLeafVector(tl_node_id),
             "some but not all treelite leaves have leaf_vector()");
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_algo");
  };
}

template <typename T, typename L>
void node2fil_dense(std::vector<dense_node>* pnodes, int root, int cur,
                    const tl::Tree<T, L>& tree, int node_id,
                    const forest_params_t& forest_params) {
  if (tree.IsLeaf(node_id)) {
    (*pnodes)[root + cur] = dense_node(val_t{.f = NAN}, NAN, 0, false, true);
    tl2fil_leaf_payload(&(*pnodes)[root + cur], tree, node_id, forest_params);
    return;
  }

  // inner node
  ASSERT(tree.SplitType(node_id) == tl::SplitFeatureType::kNumerical,
         "only numerical split nodes are supported");
  int tl_left = tree.LeftChild(node_id), tl_right = tree.RightChild(node_id);
  bool default_left = tree.DefaultLeft(node_id);
  float threshold = static_cast<float>(tree.Threshold(node_id));
  adjust_threshold(&threshold, &tl_left, &tl_right, &default_left,
                   tree.ComparisonOp(node_id));
  (*pnodes)[root + cur] = dense_node(
    val_t{.f = 0}, threshold, tree.SplitIndex(node_id), default_left, false);
  int left = 2 * cur + 1;
  node2fil_dense(pnodes, root, left, tree, tl_left, forest_params);
  node2fil_dense(pnodes, root, left + 1, tree, tl_right, forest_params);
}

template <typename T, typename L>
void tree2fil_dense(std::vector<dense_node>* pnodes, int root,
                    const tl::Tree<T, L>& tree,
                    const forest_params_t& forest_params) {
  node2fil_dense(pnodes, root, 0, tree, tree_root(tree), forest_params);
}

template <typename fil_node_t, typename T, typename L>
int tree2fil_sparse(std::vector<fil_node_t>& nodes, int root,
                    const tl::Tree<T, L>& tree,
                    const forest_params_t& forest_params) {
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  int built_index = root + 1;
  stack.push(pair_t(tree_root(tree), 0));
  while (!stack.empty()) {
    const pair_t& top = stack.top();
    int node_id = top.first;
    int cur = top.second;
    stack.pop();

    while (!tree.IsLeaf(node_id)) {
      // inner node
      ASSERT(tree.SplitType(node_id) == tl::SplitFeatureType::kNumerical,
             "only numerical split nodes are supported");
      // tl_left and tl_right are indices of the children in the treelite tree
      // (stored  as an array of nodes)
      int tl_left = tree.LeftChild(node_id),
          tl_right = tree.RightChild(node_id);
      bool default_left = tree.DefaultLeft(node_id);
      float threshold = static_cast<float>(tree.Threshold(node_id));
      adjust_threshold(&threshold, &tl_left, &tl_right, &default_left,
                       tree.ComparisonOp(node_id));

      // reserve space for child nodes
      // left is the offset of the left child node relative to the tree root
      // in the array of all nodes of the FIL sparse forest
      int left = built_index - root;
      built_index += 2;
      nodes[root + cur] =
        fil_node_t(val_t{.f = 0}, threshold, tree.SplitIndex(node_id),
                   default_left, false, left);

      // push child nodes into the stack
      stack.push(pair_t(tl_right, left + 1));
      //stack.push(pair_t(tl_left, left));
      node_id = tl_left;
      cur = left;
    }

    // leaf node
    nodes[root + cur] = fil_node_t(val_t{.f = NAN}, NAN, 0, false, true, 0);
    tl2fil_leaf_payload(&nodes[root + cur], tree, node_id, forest_params);
  }

  return root;
}

template <typename T, typename L>
size_t tl_leaf_vector_size(const tl::ModelImpl<T, L>& model) {
  const tl::Tree<T, L>& tree = model.trees[0];
  int node_key;
  for (node_key = tree_root(tree); !tree.IsLeaf(node_key);
       node_key = tree.RightChild(node_key))
    ;
  if (tree.HasLeafVector(node_key)) return tree.LeafVector(node_key).size();
  return 0;
}

// tl2fil_common is the part of conversion from a treelite model
// common for dense and sparse forests
template <typename T, typename L>
void tl2fil_common(forest_params_t* params, const tl::ModelImpl<T, L>& model,
                   const treelite_params_t* tl_params) {
  // fill in forest-indendent params
  params->algo = tl_params->algo;
  params->threshold = tl_params->threshold;

  // fill in forest-dependent params
  params->depth = max_depth(model);  // also checks for cycles

  const tl::ModelParam& param = model.param;

  // assuming either all leaves use the .leaf_vector() or all leaves use .leaf_value()
  size_t leaf_vec_size = tl_leaf_vector_size(model);
  std::string pred_transform(param.pred_transform);
  if (leaf_vec_size > 0) {
    ASSERT(leaf_vec_size == model.task_param.num_class,
           "treelite model inconsistent");
    params->num_classes = leaf_vec_size;
    params->leaf_algo = leaf_algo_t::CATEGORICAL_LEAF;

    ASSERT(tl_params->output_class,
           "output_class==true is required for multi-class models");

    ASSERT(
      pred_transform == "max_index" || pred_transform == "identity_multiclass",
      "only max_index and identity_multiclass values of pred_transform "
      "are supported for multi-class models");

  } else {
    if (model.task_param.num_class > 1) {
      params->num_classes = static_cast<int>(model.task_param.num_class);
      ASSERT(tl_params->output_class,
             "output_class==true is required for multi-class models");
      ASSERT(pred_transform == "identity_multiclass" ||
               pred_transform == "max_index" || pred_transform == "softmax" ||
               pred_transform == "multiclass_ova",
             "only identity_multiclass, max_index, multiclass_ova and softmax "
             "values of pred_transform are supported for xgboost-style "
             "multi-class classification models.");
      // this function should not know how many threads per block will be used
      params->leaf_algo = leaf_algo_t::GROVE_PER_CLASS;
    } else {
      params->num_classes = tl_params->output_class ? 2 : 1;
      ASSERT(pred_transform == "sigmoid" || pred_transform == "identity",
             "only sigmoid and identity values of pred_transform "
             "are supported for binary classification and regression models.");
      params->leaf_algo = leaf_algo_t::FLOAT_UNARY_BINARY;
    }
  }

  params->num_cols = model.num_feature;

  ASSERT(param.sigmoid_alpha == 1.0f, "sigmoid_alpha not supported");
  params->global_bias = param.global_bias;
  params->output = output_t::RAW;
  /** output_t::CLASS denotes using a threshold in FIL, when
      predict_proba == false. For all multiclass models, the best class is
      selected using argmax instead. This happens when either
      leaf_algo == CATEGORICAL_LEAF or num_classes > 2.
  **/
  if (tl_params->output_class && params->leaf_algo != CATEGORICAL_LEAF &&
      params->num_classes <= 2) {
    params->output = output_t(params->output | output_t::CLASS);
  }
  // "random forest" in treelite means tree output averaging
  if (model.average_tree_output) {
    params->output = output_t(params->output | output_t::AVG);
  }
  if (pred_transform == "sigmoid" || pred_transform == "multiclass_ova") {
    params->output = output_t(params->output | output_t::SIGMOID);
  }
  if (pred_transform == "softmax")
    params->output = output_t(params->output | output_t::SOFTMAX);
  params->num_trees = model.trees.size();
  params->blocks_per_sm = tl_params->blocks_per_sm;
}

// uses treelite model with additional tl_params to initialize FIL params
// and dense nodes (stored in *pnodes)
template <typename T, typename L>
void tl2fil_dense(std::vector<dense_node>* pnodes, forest_params_t* params,
                  const tl::ModelImpl<T, L>& model,
                  const treelite_params_t* tl_params) {
  tl2fil_common(params, model, tl_params);

  // convert the nodes
  int num_nodes = forest_num_nodes(params->num_trees, params->depth);
  pnodes->resize(num_nodes, dense_node());
  for (int i = 0; i < model.trees.size(); ++i) {
    tree2fil_dense(pnodes, i * tree_num_nodes(params->depth), model.trees[i],
                   *params);
  }
}

template <typename fil_node_t>
struct tl2fil_sparse_check_t {
  template <typename T, typename L>
  static void check(const tl::ModelImpl<T, L>& model) {
    ASSERT(false,
           "internal error: "
           "only a specialization of this template should be used");
  }
};

template <>
struct tl2fil_sparse_check_t<sparse_node16> {
  // no extra check for 16-byte sparse nodes
  template <typename T, typename L>
  static void check(const tl::ModelImpl<T, L>& model) {}
};

template <>
struct tl2fil_sparse_check_t<sparse_node8> {
  static const int MAX_FEATURES = 1 << sparse_node8::FID_NUM_BITS;
  static const int MAX_TREE_NODES = (1 << sparse_node8::LEFT_NUM_BITS) - 1;
  template <typename T, typename L>
  static void check(const tl::ModelImpl<T, L>& model) {
    // check the number of features
    int num_features = model.num_feature;
    ASSERT(num_features <= MAX_FEATURES,
           "model has %d features, "
           "but only %d supported for 8-byte sparse nodes",
           num_features, MAX_FEATURES);

    // check the number of tree nodes
    const std::vector<tl::Tree<T, L>>& trees = model.trees;
    for (int i = 0; i < trees.size(); ++i) {
      int num_nodes = trees[i].num_nodes;
      ASSERT(num_nodes <= MAX_TREE_NODES,
             "tree %d has %d nodes, "
             "but only %d supported for 8-byte sparse nodes",
             i, num_nodes, MAX_TREE_NODES);
    }
  }
};

// uses treelite model with additional tl_params to initialize FIL params,
// trees (stored in *ptrees) and sparse nodes (stored in *pnodes)
template <typename fil_node_t, typename T, typename L>
void tl2fil_sparse(std::vector<int>* ptrees, std::vector<fil_node_t>* pnodes,
                   forest_params_t* params, const tl::ModelImpl<T, L>& model,
                   const treelite_params_t* tl_params) {
  tl2fil_common(params, model, tl_params);
  tl2fil_sparse_check_t<fil_node_t>::check(model);

  size_t num_trees = model.trees.size();

  ptrees->reserve(num_trees);
  ptrees->push_back(0);
  for (size_t i = 0; i < num_trees - 1; ++i) {
    ptrees->push_back(model.trees[i].num_nodes + ptrees->back());
  }
  size_t total_nodes = ptrees->back() + model.trees.back().num_nodes;

  pnodes->resize(total_nodes);

  // convert the nodes
#pragma omp parallel for
  for (int i = 0; i < num_trees; ++i) {
    tree2fil_sparse(*pnodes, (*ptrees)[i], model.trees[i], *params);
  }

  params->num_nodes = pnodes->size();
}

void init_dense(const raft::handle_t& h, forest_t* pf, const dense_node* nodes,
                const forest_params_t* params) {
  check_params(params, true);
  dense_forest* f = new dense_forest;
  f->init(h, nodes, params);
  *pf = f;
}

template <typename fil_node_t>
void init_sparse(const raft::handle_t& h, forest_t* pf, const int* trees,
                 const fil_node_t* nodes, const forest_params_t* params) {
  check_params(params, false);
  sparse_forest<fil_node_t>* f = new sparse_forest<fil_node_t>;
  f->init(h, trees, nodes, params);
  *pf = f;
}

// explicit instantiations for init_sparse()
template void init_sparse<sparse_node16>(const raft::handle_t& h, forest_t* pf,
                                         const int* trees,
                                         const sparse_node16* nodes,
                                         const forest_params_t* params);

template void init_sparse<sparse_node8>(const raft::handle_t& h, forest_t* pf,
                                        const int* trees,
                                        const sparse_node8* nodes,
                                        const forest_params_t* params);

template <typename T, typename L>
void from_treelite(const raft::handle_t& handle, forest_t* pforest,
                   const tl::ModelImpl<T, L>& model,
                   const treelite_params_t* tl_params) {
  // Invariants on threshold and leaf types
  static_assert(std::is_same<T, float>::value || std::is_same<T, double>::value,
                "Model must contain float32 or float64 thresholds for splits");
  ASSERT((std::is_same<L, float>::value || std::is_same<L, double>::value),
         "Models with integer leaf output are not yet supported");
  // Display appropriate warnings when float64 values are being casted into
  // float32, as FIL only supports inferencing with float32 for the time being
  if (std::is_same<T, double>::value || std::is_same<L, double>::value) {
    CUML_LOG_WARN(
      "Casting all thresholds and leaf values to float32, as FIL currently "
      "doesn't support inferencing models with float64 values. "
      "This may lead to predictions with reduced accuracy.");
  }

  storage_type_t storage_type = tl_params->storage_type;
  // build dense trees by default
  if (storage_type == storage_type_t::AUTO) {
    if (tl_params->algo == algo_t::ALGO_AUTO ||
        tl_params->algo == algo_t::NAIVE) {
      int depth = max_depth(model);
      // max 2**25 dense nodes, 256 MiB dense model size
      const int LOG2_MAX_DENSE_NODES = 25;
      int log2_num_dense_nodes =
        depth + 1 + int(ceil(std::log2(model.trees.size())));
      storage_type = log2_num_dense_nodes > LOG2_MAX_DENSE_NODES
                       ? storage_type_t::SPARSE
                       : storage_type_t::DENSE;
    } else {
      // only dense storage is supported for other algorithms
      storage_type = storage_type_t::DENSE;
    }
  }

  forest_params_t params;
  switch (storage_type) {
    case storage_type_t::DENSE: {
      std::vector<dense_node> nodes;
      tl2fil_dense(&nodes, &params, model, tl_params);
      init_dense(handle, pforest, nodes.data(), &params);
      // sync is necessary as nodes is used in init_dense(),
      // but destructed at the end of this function
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      break;
    }
    case storage_type_t::SPARSE: {
      std::vector<int> trees;
      std::vector<sparse_node16> nodes;
      tl2fil_sparse(&trees, &nodes, &params, model, tl_params);
      init_sparse(handle, pforest, trees.data(), nodes.data(), &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      break;
    }
    case storage_type_t::SPARSE8: {
      std::vector<int> trees;
      std::vector<sparse_node8> nodes;
      tl2fil_sparse(&trees, &nodes, &params, model, tl_params);
      init_sparse(handle, pforest, trees.data(), nodes.data(), &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      break;
    }
    default:
      ASSERT(false, "tl_params->sparse must be one of AUTO, DENSE or SPARSE");
  }
}

void from_treelite(const raft::handle_t& handle, forest_t* pforest,
                   ModelHandle model, const treelite_params_t* tl_params) {
  const tl::Model& model_ref = *(tl::Model*)model;
  model_ref.Dispatch([&handle, pforest, tl_params](const auto& model_inner) {
    // model_inner is of the concrete type tl::ModelImpl<T, L>
    from_treelite(handle, pforest, model_inner, tl_params);
  });
}

void free(const raft::handle_t& h, forest_t f) {
  f->free(h);
  delete f;
}

void predict(const raft::handle_t& h, forest_t f, float* preds,
             const float* data, size_t num_rows, bool predict_proba) {
  f->predict(h, preds, data, num_rows, predict_proba);
}

}  // namespace fil
}  // namespace ML
