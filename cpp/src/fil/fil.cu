#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file fil.cu implements forest inference */

#include "common.cuh"

#include <cuml/fil/fil.h>
#include <cuml/fil/fnv_hash.h>
#include <cuml/common/logger.hpp>

#include <raft/cudart_utils.h>
#include <raft/handle.hpp>

#include <treelite/c_api.h>
#include <treelite/tree.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <omp.h>

#include <algorithm>
#include <bitset>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <iomanip>
#include <limits>
#include <stack>
#include <utility>

namespace ML {
namespace fil {

namespace tl = treelite;

std::ostream& operator<<(std::ostream& os, const cat_sets_owner& cso)
{
  os << "\nbits { ";
  for (uint8_t b : cso.bits) {
    os << std::bitset<BITS_PER_BYTE>(b) << " ";
  }
  os << " }\nmax_matching {";
  for (int mm : cso.max_matching) {
    os << mm << " ";
  }
  os << " }";
  return os;
}

__host__ __device__ float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

/** performs additional transformations on the array of forest predictions
    (preds) of size n; the transformations are defined by output, and include
    averaging (multiplying by inv_num_trees), adding global_bias (always done),
    sigmoid and applying threshold. in case of complement_proba,
    fills in the complement probability */
__global__ void transform_k(float* preds,
                            size_t n,
                            output_t output,
                            float inv_num_trees,
                            float threshold,
                            float global_bias,
                            bool complement_proba)
{
  size_t i = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (i >= n) return;
  if (complement_proba && i % 2 != 0) return;

  float result = preds[i];
  if ((output & output_t::AVG) != 0) result *= inv_num_trees;
  result += global_bias;
  if ((output & output_t::SIGMOID) != 0) result = sigmoid(result);
  // will not be done on CATEGORICAL_LEAF because the whole kernel will not run
  if ((output & output_t::CLASS) != 0) { result = result > threshold ? 1.0f : 0.0f; }
  // sklearn outputs numpy array in 'C' order, with the number of classes being last dimension
  // that is also the default order, so we should use the same one
  if (complement_proba) {
    preds[i]     = 1.0f - result;
    preds[i + 1] = result;
  } else
    preds[i] = result;
}

// needed to avoid expanding the dispatch template into unresolved
// compute_smem_footprint::run() calls. In infer.cu, we don't export those symbols,
// but rather one symbol for the whole template specialization, as below.
extern template int dispatch_on_fil_template_params(compute_smem_footprint, predict_params);

struct forest {
  forest(const raft::handle_t& h) : vector_leaf_(0, h.get_stream()), cat_sets_(h.get_stream()) {}

  void init_n_items(int device)
  {
    int max_shm_std = 48 * 1024;  // 48 KiB
    /// the most shared memory a kernel can request on the GPU in question
    int max_shm = 0;
    CUDA_CHECK(hipDeviceGetAttribute(&max_shm, hipDeviceAttributeSharedMemPerBlockOptin, device));
    /* Our GPUs have been growing the shared memory size generation after
       generation. Eventually, a CUDA GPU might come by that supports more
       shared memory that would fit into unsigned 16-bit int. For such a GPU,
       we would have otherwise silently overflowed the index calculation due
       to short division. It would have failed cpp tests, but we might forget
       about this source of bugs, if not for the failing assert. */
    ASSERT(max_shm < 262144,
           "internal error: please use a larger type inside"
           " infer_k for column count");
    // TODO(canonizer): use >48KiB shared memory if available
    max_shm = std::min(max_shm, max_shm_std);

    // searching for the most items per block while respecting the shared
    // memory limits creates a full linear programming problem.
    // solving it in a single equation looks less tractable than this
    for (bool predict_proba : {false, true}) {
      shmem_size_params& ssp_ = predict_proba ? proba_ssp_ : class_ssp_;
      ssp_.predict_proba      = predict_proba;
      shmem_size_params ssp   = ssp_;
      // if n_items was not provided, try from 1 to MAX_N_ITEMS. Otherwise, use as-is.
      int min_n_items = ssp.n_items == 0 ? 1 : ssp.n_items;
      int max_n_items =
        ssp.n_items == 0 ? (algo_ == algo_t::BATCH_TREE_REORG ? MAX_N_ITEMS : 1) : ssp.n_items;
      for (bool cols_in_shmem : {false, true}) {
        ssp.cols_in_shmem = cols_in_shmem;
        for (ssp.n_items = min_n_items; ssp.n_items <= max_n_items; ++ssp.n_items) {
          ssp.shm_sz = dispatch_on_fil_template_params(compute_smem_footprint(), ssp);
          if (ssp.shm_sz < max_shm) ssp_ = ssp;
        }
      }
      ASSERT(max_shm >= ssp_.shm_sz,
             "FIL out of shared memory. Perhaps the maximum number of \n"
             "supported classes is exceeded? 5'000 would still be safe.");
    }
  }

  void init_fixed_block_count(int device, int blocks_per_sm)
  {
    int max_threads_per_sm, sm_count;
    CUDA_CHECK(
      hipDeviceGetAttribute(&max_threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, device));
    blocks_per_sm = std::min(blocks_per_sm, max_threads_per_sm / FIL_TPB);
    CUDA_CHECK(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device));
    fixed_block_count_ = blocks_per_sm * sm_count;
  }

  void init_common(const raft::handle_t& h,
                   const categorical_sets& cat_sets,
                   const std::vector<float>& vector_leaf,
                   const forest_params_t* params)
  {
    depth_                           = params->depth;
    num_trees_                       = params->num_trees;
    algo_                            = params->algo;
    output_                          = params->output;
    threshold_                       = params->threshold;
    global_bias_                     = params->global_bias;
    proba_ssp_.n_items               = params->n_items;
    proba_ssp_.log2_threads_per_tree = log2(params->threads_per_tree);
    proba_ssp_.leaf_algo             = params->leaf_algo;
    proba_ssp_.num_cols              = params->num_cols;
    proba_ssp_.num_classes           = params->num_classes;
    proba_ssp_.cats_present          = cat_sets.cats_present();
    class_ssp_                       = proba_ssp_;

    int device          = h.get_device();
    hipStream_t stream = h.get_stream();
    init_n_items(device);  // n_items takes priority over blocks_per_sm
    init_fixed_block_count(device, params->blocks_per_sm);

    // vector leaf
    if (!vector_leaf.empty()) {
      vector_leaf_.resize(vector_leaf.size(), stream);

      CUDA_CHECK(hipMemcpyAsync(vector_leaf_.data(),
                                 vector_leaf.data(),
                                 vector_leaf.size() * sizeof(float),
                                 hipMemcpyHostToDevice,
                                 stream));
    }

    // categorical features
    cat_sets_ = cat_sets_device_owner(cat_sets, stream);
  }

  virtual void infer(predict_params params, hipStream_t stream) = 0;

  void predict(
    const raft::handle_t& h, float* preds, const float* data, size_t num_rows, bool predict_proba)
  {
    // Initialize prediction parameters.
    predict_params params(predict_proba ? proba_ssp_ : class_ssp_);
    params.algo     = algo_;
    params.preds    = preds;
    params.data     = data;
    params.num_rows = num_rows;
    // ignored unless predict_proba is true and algo is GROVE_PER_CLASS
    params.transform = output_;
    // fixed_block_count_ == 0 means the number of thread blocks is
    // proportional to the number of rows
    params.num_blocks = fixed_block_count_;

    /**
    The binary classification / regression (FLOAT_UNARY_BINARY) predict_proba() works as follows
      (always 2 outputs):
    RAW: output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error
    write the output of the previous stages and its complement

    The binary classification / regression (FLOAT_UNARY_BINARY) predict() works as follows
      (always 1 output):
    RAW (no values set): output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: apply threshold (equivalent to choosing best class)
    SOFTMAX is set: error

    The multi-class classification / regression (CATEGORICAL_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error

    The multi-class classification / regression (CATEGORICAL_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest probability, else output label
    0. SOFTMAX is set: error All other flags (AVG, SIGMOID, CLASS) are ignored

    The multi-class classification / regression (GROVE_PER_CLASS) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid; if SOFTMAX is also set: error
    CLASS is set: ignored
    SOFTMAX is set: softmax is applied after averaging and global_bias

    The multi-class classification / regression (GROVE_PER_CLASS) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS, SOFTMAX) are ignored

    The multi-class classification / regression (VECTOR_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid; if SOFTMAX is also set: error
    CLASS is set: ignored
    SOFTMAX is set: softmax is applied after averaging and global_bias
    All other flags (SIGMOID, CLASS, SOFTMAX) are ignored

    The multi-class classification / regression (VECTOR_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS, SOFTMAX) are ignored
    */
    output_t ot = output_;
    // Treelite applies bias before softmax, but we do after.
    // Simulating treelite order, which cancels out bias.
    // If non-proba prediction used, it still will not matter
    // for the same reason softmax will not.
    float global_bias     = (ot & output_t::SOFTMAX) != 0 ? 0.0f : global_bias_;
    bool complement_proba = false, do_transform;

    if (predict_proba) {
      // no threshold on probabilities
      ot = output_t(ot & ~output_t::CLASS);

      switch (params.leaf_algo) {
        case leaf_algo_t::FLOAT_UNARY_BINARY:
          params.num_outputs = 2;
          complement_proba   = true;
          do_transform       = true;
          break;
        case leaf_algo_t::GROVE_PER_CLASS:
          // for GROVE_PER_CLASS, averaging happens in infer_k
          ot                 = output_t(ot & ~output_t::AVG);
          params.num_outputs = params.num_classes;
          do_transform = (ot != output_t::RAW && ot != output_t::SOFTMAX) || global_bias != 0.0f;
          break;
        case leaf_algo_t::CATEGORICAL_LEAF:
          params.num_outputs = params.num_classes;
          do_transform       = ot != output_t::RAW || global_bias_ != 0.0f;
          break;
        case leaf_algo_t::VECTOR_LEAF:
          // for VECTOR_LEAF, averaging happens in infer_k
          ot                 = output_t(ot & ~output_t::AVG);
          params.num_outputs = params.num_classes;
          do_transform = (ot != output_t::RAW && ot != output_t::SOFTMAX) || global_bias != 0.0f;
          break;
        default: ASSERT(false, "internal error: predict: invalid leaf_algo %d", params.leaf_algo);
      }
    } else {
      if (params.leaf_algo == leaf_algo_t::FLOAT_UNARY_BINARY) {
        do_transform = ot != output_t::RAW || global_bias_ != 0.0f;
      } else {
        // GROVE_PER_CLASS, CATEGORICAL_LEAF: moot since choosing best class and
        // all transforms are monotonic. also, would break current code
        do_transform = false;
      }
      params.num_outputs = 1;
    }

    // Predict using the forest.
    hipStream_t stream = h.get_stream();
    infer(params, stream);

    if (do_transform) {
      size_t num_values_to_transform = (size_t)num_rows * (size_t)params.num_outputs;
      transform_k<<<raft::ceildiv(num_values_to_transform, (size_t)FIL_TPB), FIL_TPB, 0, stream>>>(
        preds,
        num_values_to_transform,
        ot,
        num_trees_ > 0 ? (1.0f / num_trees_) : 1.0f,
        threshold_,
        global_bias,
        complement_proba);
      CUDA_CHECK(hipPeekAtLastError());
    }
  }

  virtual void free(const raft::handle_t& h)
  {
    cat_sets_.release();
    vector_leaf_.release();
  }

  virtual ~forest() {}

  int num_trees_     = 0;
  int depth_         = 0;
  algo_t algo_       = algo_t::NAIVE;
  output_t output_   = output_t::RAW;
  float threshold_   = 0.5;
  float global_bias_ = 0;
  shmem_size_params class_ssp_, proba_ssp_;
  int fixed_block_count_ = 0;
  // Optionally used
  rmm::device_uvector<float> vector_leaf_;
  cat_sets_device_owner cat_sets_;
};

struct dense_forest : forest {
  dense_forest(const raft::handle_t& h) : forest(h), nodes_(0, h.get_stream()) {}

  void transform_trees(const dense_node* nodes)
  {
    /* Populate node information:
       For each tree, the nodes are still stored in the breadth-first,
       left-to-right order. However, instead of storing the nodes of the same
       tree adjacently, it uses a different layout. In this layout, the roots
       of all trees (node 0) are stored first, followed by left children of
       the roots of all trees (node 1), followed by the right children of the
       roots of all trees (node 2), and so on.
    */
    int global_node = 0;
    for (int tree = 0; tree < num_trees_; ++tree) {
      int tree_node = 0;
      // the counters `level` and `branch` are not used for computing node
      // indices, they are only here to highlight the node ordering within
      // each tree
      for (int level = 0; level <= depth_; ++level) {
        for (int branch = 0; branch < 1 << level; ++branch) {
          h_nodes_[tree_node * num_trees_ + tree] = nodes[global_node];
          ++tree_node;
          ++global_node;
        }
      }
    }
  }

  void init(const raft::handle_t& h,
            const categorical_sets& cat_sets,
            const std::vector<float>& vector_leaf,
            const dense_node* nodes,
            const forest_params_t* params)
  {
    init_common(h, cat_sets, vector_leaf, params);
    if (algo_ == algo_t::NAIVE) algo_ = algo_t::BATCH_TREE_REORG;

    int num_nodes = forest_num_nodes(num_trees_, depth_);
    nodes_.resize(num_nodes, h.get_stream());
    h_nodes_.resize(num_nodes);
    if (algo_ == algo_t::NAIVE) {
      std::copy(nodes, nodes + num_nodes, h_nodes_.begin());
    } else {
      transform_trees(nodes);
    }
    CUDA_CHECK(hipMemcpyAsync(nodes_.data(),
                               h_nodes_.data(),
                               num_nodes * sizeof(dense_node),
                               hipMemcpyHostToDevice,
                               h.get_stream()));
    // copy must be finished before freeing the host data
    CUDA_CHECK(hipStreamSynchronize(h.get_stream()));
    h_nodes_.clear();
    h_nodes_.shrink_to_fit();
  }

  virtual void infer(predict_params params, hipStream_t stream) override
  {
    dense_storage forest(cat_sets_.accessor(),
                         vector_leaf_.data(),
                         nodes_.data(),
                         num_trees_,
                         algo_ == algo_t::NAIVE ? tree_num_nodes(depth_) : 1,
                         algo_ == algo_t::NAIVE ? 1 : num_trees_);
    fil::infer(forest, params, stream);
  }

  virtual void free(const raft::handle_t& h) override
  {
    nodes_.release();
    forest::free(h);
  }

  rmm::device_uvector<dense_node> nodes_;
  thrust::host_vector<dense_node> h_nodes_;
};

template <typename node_t>
struct sparse_forest : forest {
  sparse_forest(const raft::handle_t& h)
    : forest(h), trees_(0, h.get_stream()), nodes_(0, h.get_stream())
  {
  }

  void init(const raft::handle_t& h,
            const categorical_sets& cat_sets,
            const std::vector<float>& vector_leaf,
            const int* trees,
            const node_t* nodes,
            const forest_params_t* params)
  {
    init_common(h, cat_sets, vector_leaf, params);
    if (algo_ == algo_t::ALGO_AUTO) algo_ = algo_t::NAIVE;
    depth_     = 0;  // a placeholder value
    num_nodes_ = params->num_nodes;

    // trees
    trees_.resize(num_trees_, h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(
      trees_.data(), trees, sizeof(int) * num_trees_, hipMemcpyHostToDevice, h.get_stream()));

    // nodes
    nodes_.resize(num_nodes_, h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(
      nodes_.data(), nodes, sizeof(node_t) * num_nodes_, hipMemcpyHostToDevice, h.get_stream()));
  }

  virtual void infer(predict_params params, hipStream_t stream) override
  {
    sparse_storage<node_t> forest(
      cat_sets_.accessor(), vector_leaf_.data(), trees_.data(), nodes_.data(), num_trees_);
    fil::infer(forest, params, stream);
  }

  void free(const raft::handle_t& h) override
  {
    forest::free(h);
    trees_.release();
    nodes_.release();
  }

  int num_nodes_ = 0;
  rmm::device_uvector<int> trees_;
  rmm::device_uvector<node_t> nodes_;
};

void check_params(const forest_params_t* params, bool dense)
{
  if (dense) {
    ASSERT(params->depth >= 0, "depth must be non-negative for dense forests");
  } else {
    ASSERT(params->num_nodes >= 0, "num_nodes must be non-negative for sparse forests");
    ASSERT(params->algo == algo_t::NAIVE || params->algo == algo_t::ALGO_AUTO,
           "only ALGO_AUTO and NAIVE algorithms are supported "
           "for sparse forests");
  }
  ASSERT(params->num_trees >= 0, "num_trees must be non-negative");
  ASSERT(params->num_cols >= 0, "num_cols must be non-negative");
  switch (params->algo) {
    case algo_t::ALGO_AUTO:
    case algo_t::NAIVE:
    case algo_t::TREE_REORG:
    case algo_t::BATCH_TREE_REORG: break;
    default: ASSERT(false, "algo should be ALGO_AUTO, NAIVE, TREE_REORG or BATCH_TREE_REORG");
  }
  switch (params->leaf_algo) {
    case leaf_algo_t::FLOAT_UNARY_BINARY:
      if ((params->output & output_t::CLASS) != 0) {
        ASSERT(params->num_classes == 2,
               "only supporting binary"
               " classification using FLOAT_UNARY_BINARY");
      } else {
        ASSERT(params->num_classes == 1,
               "num_classes must be 1 for "
               "regression");
      }
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax does not make sense for leaf_algo == FLOAT_UNARY_BINARY");
      break;
    case leaf_algo_t::GROVE_PER_CLASS:
      ASSERT(params->threads_per_tree == 1, "multiclass not supported with threads_per_tree > 1");
      ASSERT(params->num_classes > 2,
             "num_classes > 2 is required for leaf_algo == GROVE_PER_CLASS");
      ASSERT(params->num_trees % params->num_classes == 0,
             "num_classes must divide num_trees evenly for GROVE_PER_CLASS");
      break;
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(params->threads_per_tree == 1, "multiclass not supported with threads_per_tree > 1");
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == CATEGORICAL_LEAF");
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax not supported for leaf_algo == CATEGORICAL_LEAF");
      break;
    case leaf_algo_t::VECTOR_LEAF:
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == VECTOR_LEAF");
      break;
    default:
      ASSERT(false,
             "leaf_algo must be FLOAT_UNARY_BINARY, CATEGORICAL_LEAF"
             " or GROVE_PER_CLASS");
  }
  // output_t::RAW == 0, and doesn't have a separate flag
  if ((params->output & ~output_t::ALL_SET) != 0) {
    ASSERT(false, "output should be a combination of RAW, AVG, SIGMOID, CLASS and SOFTMAX");
  }
  ASSERT(~params->output & (output_t::SIGMOID | output_t::SOFTMAX),
         "combining softmax and sigmoid is not supported");
  ASSERT(params->blocks_per_sm >= 0, "blocks_per_sm must be nonnegative");
  ASSERT(params->n_items >= 0, "n_items must be non-negative");
  ASSERT(params->threads_per_tree > 0, "threads_per_tree must be positive");
  ASSERT(thrust::detail::is_power_of_2(params->threads_per_tree),
         "threads_per_tree must be a power of 2");
  ASSERT(params->threads_per_tree <= FIL_TPB,
         "threads_per_tree must not "
         "exceed block size %d",
         FIL_TPB);
}

template <typename T, typename L>
int tree_root(const tl::Tree<T, L>& tree)
{
  return 0;  // Treelite format assumes that the root is 0
}

template <typename T, typename L>
inline int max_depth(const tl::Tree<T, L>& tree)
{
  // trees of this depth aren't used, so it most likely means bad input data,
  // e.g. cycles in the forest
  const int DEPTH_LIMIT = 500;
  int root_index        = tree_root(tree);
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  stack.push(pair_t(root_index, 0));
  int max_depth = 0;
  while (!stack.empty()) {
    const pair_t& pair = stack.top();
    int node_id        = pair.first;
    int depth          = pair.second;
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      stack.push(pair_t(tree.LeftChild(node_id), depth + 1));
      node_id = tree.RightChild(node_id);
      depth++;
      ASSERT(depth < DEPTH_LIMIT, "depth limit reached, might be a cycle in the tree");
    }
    // only need to update depth for leaves
    max_depth = std::max(max_depth, depth);
  }
  return max_depth;
}

template <typename T, typename L>
int max_depth(const tl::ModelImpl<T, L>& model)
{
  int depth         = 0;
  const auto& trees = model.trees;
#pragma omp parallel for reduction(max : depth)
  for (size_t i = 0; i < trees.size(); ++i) {
    const auto& tree = trees[i];
    depth            = std::max(depth, max_depth(tree));
  }
  return depth;
}

void elementwise_combine(std::vector<cat_feature_counters>& dst,
                         const std::vector<cat_feature_counters>& extra)
{
  std::transform(dst.begin(), dst.end(), extra.begin(), dst.begin(), cat_feature_counters::combine);
}

// constructs a vector of size n_cols (number of features, or columns) from a Treelite tree,
// where each feature has a maximum matching category and node count (from this tree alone).
template <typename T, typename L>
inline std::vector<cat_feature_counters> cat_counter_vec(const tl::Tree<T, L>& tree, int n_cols)
{
  std::vector<cat_feature_counters> res(n_cols);
  std::stack<int> stack;
  stack.push(tree_root(tree));
  while (!stack.empty()) {
    int node_id = stack.top();
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      if (tree.SplitType(node_id) == tl::SplitFeatureType::kCategorical) {
        std::vector<std::uint32_t> mmv = tree.MatchingCategories(node_id);
        int max_matching_cat;
        if (mmv.size() > 0) {
          // in `struct cat_feature_counters` and GPU structures, max matching category is an int
          // cast is safe because all precise int floats fit into ints, which are asserted to be 32
          // bits
          max_matching_cat = mmv.back();
          ASSERT(max_matching_cat <= MAX_PRECISE_INT_FLOAT,
                 "FIL cannot infer on "
                 "more than %d matching categories",
                 MAX_PRECISE_INT_FLOAT);
        } else {
          max_matching_cat = -1;
        }
        cat_feature_counters& counters = res[tree.SplitIndex(node_id)];
        counters =
          cat_feature_counters::combine(counters, cat_feature_counters{max_matching_cat, 1});
      }
      stack.push(tree.LeftChild(node_id));
      node_id = tree.RightChild(node_id);
    }
  }
  return res;
}

// computes overall categorical bit pool size for a tree imported from the Treelite tree
template <typename T, typename L>
inline std::size_t bit_pool_size(const tl::Tree<T, L>& tree, const categorical_sets& cat_sets)
{
  std::size_t size = 0;
  std::stack<int> stack;
  stack.push(tree_root(tree));
  while (!stack.empty()) {
    int node_id = stack.top();
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      if (tree.SplitType(node_id) == tl::SplitFeatureType::kCategorical) {
        int fid = tree.SplitIndex(node_id);
        size += cat_sets.sizeof_mask(fid);
      }
      stack.push(tree.LeftChild(node_id));
      node_id = tree.RightChild(node_id);
    }
  }
  return size;
}

template <typename T, typename L>
cat_sets_owner allocate_cat_sets_owner(const tl::ModelImpl<T, L>& model)
{
#pragma omp declare reduction(cat_counter_vec_red : std::vector<cat_feature_counters> \
      : elementwise_combine(omp_out, omp_in))                 \
    initializer(omp_priv = omp_orig)
  const auto& trees = model.trees;
  cat_sets_owner cat_sets;
  std::vector<cat_feature_counters> counters(model.num_feature);
#pragma omp parallel for reduction(cat_counter_vec_red : counters)
  for (std::size_t i = 0; i < trees.size(); ++i) {
    elementwise_combine(counters, cat_counter_vec(trees[i], model.num_feature));
  }
  cat_sets.consume_counters(counters);
  std::vector<std::size_t> bit_pool_sizes(trees.size());
#pragma omp parallel for
  for (std::size_t i = 0; i < trees.size(); ++i) {
    bit_pool_sizes[i] = bit_pool_size(trees[i], cat_sets.accessor());
  }
  cat_sets.consume_bit_pool_sizes(bit_pool_sizes);
  return cat_sets;
}

void adjust_threshold(
  float* pthreshold, int* tl_left, int* tl_right, bool* default_left, tl::Operator comparison_op)
{
  // in treelite (take left node if val [op] threshold),
  // the meaning of the condition is reversed compared to FIL;
  // thus, "<" in treelite corresonds to comparison ">=" used by FIL
  // https://github.com/dmlc/treelite/blob/master/include/treelite/tree.h#L243
  if (isnan(*pthreshold)) {
    std::swap(*tl_left, *tl_right);
    *default_left = !*default_left;
    return;
  }
  switch (comparison_op) {
    case tl::Operator::kLT: break;
    case tl::Operator::kLE:
      // x <= y is equivalent to x < y', where y' is the next representable float
      *pthreshold = std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
      break;
    case tl::Operator::kGT:
      // x > y is equivalent to x >= y', where y' is the next representable float
      // left and right still need to be swapped
      *pthreshold = std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
    case tl::Operator::kGE:
      // swap left and right
      std::swap(*tl_left, *tl_right);
      *default_left = !*default_left;
      break;
    default: ASSERT(false, "only <, >, <= and >= comparisons are supported");
  }
}

/** if the vector consists of zeros and a single one, return the position
for the one (assumed class label). Else, asserts false.
If the vector contains a NAN, asserts false */
template <typename L>
int find_class_label_from_one_hot(L* vector, int len)
{
  bool found_label = false;
  int out;
  for (int i = 0; i < len; ++i) {
    if (vector[i] == static_cast<L>(1.0)) {
      ASSERT(!found_label, "label vector contains multiple 1.0f");
      out         = i;
      found_label = true;
    } else {
      ASSERT(vector[i] == static_cast<L>(0.0),
             "label vector contains values other than 0.0 and 1.0");
    }
  }
  ASSERT(found_label, "did not find 1.0f in vector");
  return out;
}

template <typename fil_node_t, typename T, typename L>
void tl2fil_leaf_payload(fil_node_t* fil_node,
                         int fil_node_id,
                         const tl::Tree<T, L>& tl_tree,
                         int tl_node_id,
                         const forest_params_t& forest_params,
                         std::vector<float>* vector_leaf,
                         size_t* leaf_counter)
{
  auto vec = tl_tree.LeafVector(tl_node_id);
  switch (forest_params.leaf_algo) {
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(vec.size() == static_cast<std::size_t>(forest_params.num_classes),
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = find_class_label_from_one_hot(&vec[0], vec.size());
      break;
    case leaf_algo_t::VECTOR_LEAF: {
      ASSERT(vec.size() == static_cast<std::size_t>(forest_params.num_classes),
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = *leaf_counter;
      for (int k = 0; k < forest_params.num_classes; k++) {
        (*vector_leaf)[*leaf_counter * forest_params.num_classes + k] = vec[k];
      }
      (*leaf_counter)++;
      break;
    }
    case leaf_algo_t::FLOAT_UNARY_BINARY:
    case leaf_algo_t::GROVE_PER_CLASS:
      fil_node->val.f = static_cast<float>(tl_tree.LeafValue(tl_node_id));
      ASSERT(!tl_tree.HasLeafVector(tl_node_id),
             "some but not all treelite leaves have leaf_vector()");
      break;
    default: ASSERT(false, "internal error: invalid leaf_algo");
  };
}

template <typename fil_node_t>
struct conversion_state {
  fil_node_t node;
  int tl_left;
  int tl_right;
};

// modifies cat_sets
template <typename fil_node_t, typename T, typename L>
conversion_state<fil_node_t> tl2fil_inner_node(int fil_left_child,
                                               const tl::Tree<T, L>& tree,
                                               int tl_node_id,
                                               const forest_params_t& forest_params,
                                               cat_sets_owner* cat_sets,
                                               std::size_t* bit_pool_offset)
{
  int tl_left = tree.LeftChild(tl_node_id), tl_right = tree.RightChild(tl_node_id);
  val_t split         = {.f = NAN};  // yes there's a default initializer already
  int feature_id      = tree.SplitIndex(tl_node_id);
  bool is_categorical = tree.SplitType(tl_node_id) == tl::SplitFeatureType::kCategorical;
  bool default_left   = tree.DefaultLeft(tl_node_id);
  if (tree.SplitType(tl_node_id) == tl::SplitFeatureType::kNumerical) {
    split.f = static_cast<float>(tree.Threshold(tl_node_id));
    adjust_threshold(&split.f, &tl_left, &tl_right, &default_left, tree.ComparisonOp(tl_node_id));
  } else if (tree.SplitType(tl_node_id) == tl::SplitFeatureType::kCategorical) {
    // for FIL, the list of categories is always for the right child
    if (!tree.CategoriesListRightChild(tl_node_id)) {
      std::swap(tl_left, tl_right);
      default_left = !default_left;
    }
    int sizeof_mask = cat_sets->accessor().sizeof_mask(feature_id);
    split.idx       = *bit_pool_offset;
    *bit_pool_offset += sizeof_mask;
    // cat_sets->bits have been zero-initialized
    uint8_t* bits = &cat_sets->bits[split.idx];
    for (std::uint32_t category : tree.MatchingCategories(tl_node_id)) {
      bits[category / BITS_PER_BYTE] |= 1 << (category % BITS_PER_BYTE);
    }
  } else {
    ASSERT(false, "only numerical and categorical split nodes are supported");
  }
  fil_node_t node;
  if constexpr (std::is_same<fil_node_t, dense_node>()) {
    node = fil_node_t({}, split, feature_id, default_left, false, is_categorical);
  } else {
    node = fil_node_t({}, split, feature_id, default_left, false, is_categorical, fil_left_child);
  }
  return conversion_state<fil_node_t>{node, tl_left, tl_right};
}

template <typename T, typename L>
void node2fil_dense(std::vector<dense_node>* pnodes,
                    int root,
                    int cur,
                    const tl::Tree<T, L>& tree,
                    int node_id,
                    const forest_params_t& forest_params,
                    std::vector<float>* vector_leaf,
                    std::size_t* leaf_counter,
                    cat_sets_owner* cat_sets,
                    std::size_t* bit_pool_offset)
{
  if (tree.IsLeaf(node_id)) {
    (*pnodes)[root + cur] = dense_node({}, {}, 0, false, true, false);
    tl2fil_leaf_payload(
      &(*pnodes)[root + cur], root + cur, tree, node_id, forest_params, vector_leaf, leaf_counter);
    return;
  }

  // inner node
  int left = 2 * cur + 1;
  conversion_state<dense_node> cs =
    tl2fil_inner_node<dense_node>(left, tree, node_id, forest_params, cat_sets, bit_pool_offset);
  (*pnodes)[root + cur] = cs.node;
  node2fil_dense(pnodes,
                 root,
                 left,
                 tree,
                 cs.tl_left,
                 forest_params,
                 vector_leaf,
                 leaf_counter,
                 cat_sets,
                 bit_pool_offset);
  node2fil_dense(pnodes,
                 root,
                 left + 1,
                 tree,
                 cs.tl_right,
                 forest_params,
                 vector_leaf,
                 leaf_counter,
                 cat_sets,
                 bit_pool_offset);
}

template <typename T, typename L>
void tree2fil_dense(std::vector<dense_node>* pnodes,
                    int root,
                    const tl::Tree<T, L>& tree,
                    std::size_t tree_idx,
                    const forest_params_t& forest_params,
                    std::vector<float>* vector_leaf,
                    std::size_t* leaf_counter,
                    cat_sets_owner* cat_sets)
{
  node2fil_dense(pnodes,
                 root,
                 0,
                 tree,
                 tree_root(tree),
                 forest_params,
                 vector_leaf,
                 leaf_counter,
                 cat_sets,
                 &cat_sets->bit_pool_offsets[tree_idx]);
}

template <typename fil_node_t, typename T, typename L>
int tree2fil_sparse(std::vector<fil_node_t>& nodes,
                    int root,
                    const tl::Tree<T, L>& tree,
                    std::size_t tree_idx,
                    const forest_params_t& forest_params,
                    std::vector<float>* vector_leaf,
                    std::size_t* leaf_counter,
                    cat_sets_owner* cat_sets)
{
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  int built_index = root + 1;
  stack.push(pair_t(tree_root(tree), 0));
  while (!stack.empty()) {
    const pair_t& top = stack.top();
    int node_id       = top.first;
    int cur           = top.second;
    stack.pop();

    while (!tree.IsLeaf(node_id)) {
      // reserve space for child nodes
      // left is the offset of the left child node relative to the tree root
      // in the array of all nodes of the FIL sparse forest
      int left = built_index - root;
      built_index += 2;
      conversion_state<fil_node_t> cs = tl2fil_inner_node<fil_node_t>(
        left, tree, node_id, forest_params, cat_sets, &cat_sets->bit_pool_offsets[tree_idx]);
      nodes[root + cur] = cs.node;
      // push child nodes into the stack
      stack.push(pair_t(cs.tl_right, left + 1));
      // stack.push(pair_t(tl_left, left));
      node_id = cs.tl_left;
      cur     = left;
    }

    // leaf node
    nodes[root + cur] = fil_node_t({}, {}, 0, false, true, false, 0);
    tl2fil_leaf_payload(
      &nodes[root + cur], root + cur, tree, node_id, forest_params, vector_leaf, leaf_counter);
  }

  return root;
}

struct level_entry {
  int n_branch_nodes, n_leaves;
};
typedef std::pair<int, int> pair_t;
// hist has branch and leaf count given depth
template <typename T, typename L>
inline void tree_depth_hist(const tl::Tree<T, L>& tree, std::vector<level_entry>& hist)
{
  std::stack<pair_t> stack;  // {tl_id, depth}
  stack.push({tree_root(tree), 0});
  while (!stack.empty()) {
    const pair_t& top = stack.top();
    int node_id       = top.first;
    int depth         = top.second;
    stack.pop();

    while (!tree.IsLeaf(node_id)) {
      if (static_cast<std::size_t>(depth) >= hist.size()) hist.resize(depth + 1, {0, 0});
      hist[depth].n_branch_nodes++;
      stack.push({tree.LeftChild(node_id), depth + 1});
      node_id = tree.RightChild(node_id);
      depth++;
    }

    if (static_cast<std::size_t>(depth) >= hist.size()) hist.resize(depth + 1, {0, 0});
    hist[depth].n_leaves++;
  }
}

template <typename T, typename L>
std::stringstream depth_hist_and_max(const tl::ModelImpl<T, L>& model)
{
  using namespace std;
  vector<level_entry> hist;
  for (const auto& tree : model.trees)
    tree_depth_hist(tree, hist);

  int min_leaf_depth = -1, leaves_times_depth = 0, total_branches = 0, total_leaves = 0;
  stringstream forest_shape;
  ios default_state(nullptr);
  default_state.copyfmt(forest_shape);
  forest_shape << "Depth histogram:" << endl << "depth branches leaves   nodes" << endl;
  for (std::size_t level = 0; level < hist.size(); ++level) {
    level_entry e = hist[level];
    forest_shape << setw(5) << level << setw(9) << e.n_branch_nodes << setw(7) << e.n_leaves
                 << setw(8) << e.n_branch_nodes + e.n_leaves << endl;
    forest_shape.copyfmt(default_state);
    if (e.n_leaves && min_leaf_depth == -1) min_leaf_depth = level;
    leaves_times_depth += e.n_leaves * level;
    total_branches += e.n_branch_nodes;
    total_leaves += e.n_leaves;
  }
  int total_nodes = total_branches + total_leaves;
  forest_shape << "Total: branches: " << total_branches << " leaves: " << total_leaves
               << " nodes: " << total_nodes << endl;
  forest_shape << "Avg nodes per tree: " << setprecision(2)
               << total_nodes / (float)hist[0].n_branch_nodes << endl;
  forest_shape.copyfmt(default_state);
  forest_shape << "Leaf depth: min: " << min_leaf_depth << " avg: " << setprecision(2) << fixed
               << leaves_times_depth / (float)total_leaves << " max: " << hist.size() - 1 << endl;
  forest_shape.copyfmt(default_state);

  vector<char> hist_bytes(hist.size() * sizeof(hist[0]));
  memcpy(&hist_bytes[0], &hist[0], hist_bytes.size());
  // std::hash does not promise to not be identity. Xoring plain numbers which
  // add up to one another erases information, hence, std::hash is unsuitable here
  forest_shape << "Depth histogram fingerprint: " << hex
               << fowler_noll_vo_fingerprint64_32(hist_bytes.begin(), hist_bytes.end()) << endl;
  forest_shape.copyfmt(default_state);

  return forest_shape;
}

template <typename T, typename L>
size_t tl_leaf_vector_size(const tl::ModelImpl<T, L>& model)
{
  const tl::Tree<T, L>& tree = model.trees[0];
  int node_key;
  for (node_key = tree_root(tree); !tree.IsLeaf(node_key); node_key = tree.RightChild(node_key))
    ;
  if (tree.HasLeafVector(node_key)) return tree.LeafVector(node_key).size();
  return 0;
}

// tl2fil_common is the part of conversion from a treelite model
// common for dense and sparse forests
template <typename T, typename L>
void tl2fil_common(forest_params_t* params,
                   const tl::ModelImpl<T, L>& model,
                   const treelite_params_t* tl_params)
{
  // fill in forest-indendent params
  params->algo      = tl_params->algo;
  params->threshold = tl_params->threshold;

  // fill in forest-dependent params
  params->depth = max_depth(model);  // also checks for cycles

  const tl::ModelParam& param = model.param;

  // assuming either all leaves use the .leaf_vector() or all leaves use .leaf_value()
  size_t leaf_vec_size = tl_leaf_vector_size(model);
  std::string pred_transform(param.pred_transform);
  if (leaf_vec_size > 0) {
    ASSERT(leaf_vec_size == model.task_param.num_class, "treelite model inconsistent");
    params->num_classes = leaf_vec_size;
    params->leaf_algo   = leaf_algo_t::VECTOR_LEAF;

    ASSERT(pred_transform == "max_index" || pred_transform == "identity_multiclass",
           "only max_index and identity_multiclass values of pred_transform "
           "are supported for multi-class models");

  } else {
    if (model.task_param.num_class > 1) {
      params->num_classes = static_cast<int>(model.task_param.num_class);
      ASSERT(tl_params->output_class, "output_class==true is required for multi-class models");
      ASSERT(pred_transform == "identity_multiclass" || pred_transform == "max_index" ||
               pred_transform == "softmax" || pred_transform == "multiclass_ova",
             "only identity_multiclass, max_index, multiclass_ova and softmax "
             "values of pred_transform are supported for xgboost-style "
             "multi-class classification models.");
      // this function should not know how many threads per block will be used
      params->leaf_algo = leaf_algo_t::GROVE_PER_CLASS;
    } else {
      params->num_classes = tl_params->output_class ? 2 : 1;
      ASSERT(pred_transform == "sigmoid" || pred_transform == "identity",
             "only sigmoid and identity values of pred_transform "
             "are supported for binary classification and regression models.");
      params->leaf_algo = leaf_algo_t::FLOAT_UNARY_BINARY;
    }
  }

  params->num_cols = model.num_feature;

  ASSERT(param.sigmoid_alpha == 1.0f, "sigmoid_alpha not supported");
  params->global_bias = param.global_bias;
  params->output      = output_t::RAW;
  /** output_t::CLASS denotes using a threshold in FIL, when
      predict_proba == false. For all multiclass models, the best class is
      selected using argmax instead. This happens when either
      leaf_algo == CATEGORICAL_LEAF or num_classes > 2.
  **/
  if (tl_params->output_class && params->leaf_algo != CATEGORICAL_LEAF &&
      params->num_classes <= 2) {
    params->output = output_t(params->output | output_t::CLASS);
  }
  // "random forest" in treelite means tree output averaging
  if (model.average_tree_output) { params->output = output_t(params->output | output_t::AVG); }
  if (pred_transform == "sigmoid" || pred_transform == "multiclass_ova") {
    params->output = output_t(params->output | output_t::SIGMOID);
  }
  if (pred_transform == "softmax") params->output = output_t(params->output | output_t::SOFTMAX);
  params->num_trees        = model.trees.size();
  params->blocks_per_sm    = tl_params->blocks_per_sm;
  params->threads_per_tree = tl_params->threads_per_tree;
  params->n_items          = tl_params->n_items;
}

// uses treelite model with additional tl_params to initialize FIL params
// and dense nodes (stored in *pnodes)
template <typename threshold_t, typename leaf_t>
void tl2fil_dense(std::vector<dense_node>* pnodes,
                  forest_params_t* params,
                  const tl::ModelImpl<threshold_t, leaf_t>& model,
                  const treelite_params_t* tl_params,
                  cat_sets_owner* cat_sets,
                  std::vector<float>* vector_leaf)
{
  tl2fil_common(params, model, tl_params);

  // convert the nodes
  int num_nodes           = forest_num_nodes(params->num_trees, params->depth);
  int max_leaves_per_tree = (tree_num_nodes(params->depth) + 1) / 2;
  if (params->leaf_algo == VECTOR_LEAF) {
    vector_leaf->resize(max_leaves_per_tree * params->num_trees * params->num_classes);
  }
  *cat_sets = allocate_cat_sets_owner(model);
  pnodes->resize(num_nodes, dense_node());
  for (std::size_t i = 0; i < model.trees.size(); ++i) {
    size_t leaf_counter = max_leaves_per_tree * i;
    tree2fil_dense(pnodes,
                   i * tree_num_nodes(params->depth),
                   model.trees[i],
                   i,
                   *params,
                   vector_leaf,
                   &leaf_counter,
                   cat_sets);
  }
}

template <typename fil_node_t>
struct tl2fil_sparse_check_t {
  template <typename threshold_t, typename leaf_t>
  static void check(const tl::ModelImpl<threshold_t, leaf_t>& model)
  {
    ASSERT(false,
           "internal error: "
           "only a specialization of this template should be used");
  }
};

template <>
struct tl2fil_sparse_check_t<sparse_node16> {
  // no extra check for 16-byte sparse nodes
  template <typename threshold_t, typename leaf_t>
  static void check(const tl::ModelImpl<threshold_t, leaf_t>& model)
  {
  }
};

template <>
struct tl2fil_sparse_check_t<sparse_node8> {
  static const int MAX_FEATURES   = 1 << sparse_node8::FID_NUM_BITS;
  static const int MAX_TREE_NODES = (1 << sparse_node8::LEFT_NUM_BITS) - 1;
  template <typename threshold_t, typename leaf_t>
  static void check(const tl::ModelImpl<threshold_t, leaf_t>& model)
  {
    // check the number of features
    int num_features = model.num_feature;
    ASSERT(num_features <= MAX_FEATURES,
           "model has %d features, "
           "but only %d supported for 8-byte sparse nodes",
           num_features,
           MAX_FEATURES);

    // check the number of tree nodes
    const std::vector<tl::Tree<threshold_t, leaf_t>>& trees = model.trees;
    for (std::size_t i = 0; i < trees.size(); ++i) {
      int num_nodes = trees[i].num_nodes;
      ASSERT(num_nodes <= MAX_TREE_NODES,
             "tree %zu has %d nodes, "
             "but only %d supported for 8-byte sparse nodes",
             i,
             num_nodes,
             MAX_TREE_NODES);
    }
  }
};

// uses treelite model with additional tl_params to initialize FIL params,
// trees (stored in *ptrees) and sparse nodes (stored in *pnodes)
template <typename fil_node_t, typename threshold_t, typename leaf_t>
void tl2fil_sparse(std::vector<int>* ptrees,
                   std::vector<fil_node_t>* pnodes,
                   forest_params_t* params,
                   const tl::ModelImpl<threshold_t, leaf_t>& model,
                   const treelite_params_t* tl_params,
                   cat_sets_owner* cat_sets,
                   std::vector<float>* vector_leaf)
{
  tl2fil_common(params, model, tl_params);
  tl2fil_sparse_check_t<fil_node_t>::check(model);

  size_t num_trees = model.trees.size();

  ptrees->reserve(num_trees);
  ptrees->push_back(0);
  for (size_t i = 0; i < num_trees - 1; ++i) {
    ptrees->push_back(model.trees[i].num_nodes + ptrees->back());
  }
  size_t total_nodes = ptrees->back() + model.trees.back().num_nodes;

  if (params->leaf_algo == VECTOR_LEAF) {
    size_t max_leaves = (total_nodes + num_trees) / 2;
    vector_leaf->resize(max_leaves * params->num_classes);
  }

  *cat_sets = allocate_cat_sets_owner(model);
  pnodes->resize(total_nodes);

// convert the nodes
#pragma omp parallel for
  for (std::size_t i = 0; i < num_trees; ++i) {
    // Max number of leaves processed so far
    size_t leaf_counter = ((*ptrees)[i] + i) / 2;
    tree2fil_sparse(
      *pnodes, (*ptrees)[i], model.trees[i], i, *params, vector_leaf, &leaf_counter, cat_sets);
  }

  params->num_nodes = pnodes->size();
}

void init_dense(const raft::handle_t& h,
                forest_t* pf,
                const categorical_sets& cat_sets,
                const std::vector<float>& vector_leaf,
                const dense_node* nodes,
                const forest_params_t* params)
{
  check_params(params, true);
  dense_forest* f = new dense_forest(h);
  f->init(h, cat_sets, vector_leaf, nodes, params);
  *pf = f;
}

template <typename fil_node_t>
void init_sparse(const raft::handle_t& h,
                 forest_t* pf,
                 const categorical_sets& cat_sets,
                 const std::vector<float>& vector_leaf,
                 const int* trees,
                 const fil_node_t* nodes,
                 const forest_params_t* params)
{
  check_params(params, false);
  sparse_forest<fil_node_t>* f = new sparse_forest<fil_node_t>(h);
  f->init(h, cat_sets, vector_leaf, trees, nodes, params);
  *pf = f;
}

// explicit instantiations for init_sparse()
template void init_sparse<sparse_node16>(const raft::handle_t& h,
                                         forest_t* pf,
                                         const categorical_sets& cat_sets,
                                         const std::vector<float>& vector_leaf,
                                         const int* trees,
                                         const sparse_node16* nodes,
                                         const forest_params_t* params);

template void init_sparse<sparse_node8>(const raft::handle_t& h,
                                        forest_t* pf,
                                        const categorical_sets& cat_sets,
                                        const std::vector<float>& vector_leaf,
                                        const int* trees,
                                        const sparse_node8* nodes,
                                        const forest_params_t* params);

template <typename threshold_t, typename leaf_t>
void from_treelite(const raft::handle_t& handle,
                   forest_t* pforest,
                   const tl::ModelImpl<threshold_t, leaf_t>& model,
                   const treelite_params_t* tl_params)
{
  // Invariants on threshold and leaf types
  static_assert(std::is_same<threshold_t, float>::value || std::is_same<threshold_t, double>::value,
                "Model must contain float32 or float64 thresholds for splits");
  ASSERT((std::is_same<leaf_t, float>::value || std::is_same<leaf_t, double>::value),
         "Models with integer leaf output are not yet supported");
  // Display appropriate warnings when float64 values are being casted into
  // float32, as FIL only supports inferencing with float32 for the time being
  if (std::is_same<threshold_t, double>::value || std::is_same<leaf_t, double>::value) {
    CUML_LOG_WARN(
      "Casting all thresholds and leaf values to float32, as FIL currently "
      "doesn't support inferencing models with float64 values. "
      "This may lead to predictions with reduced accuracy.");
  }

  storage_type_t storage_type = tl_params->storage_type;
  // build dense trees by default
  if (storage_type == storage_type_t::AUTO) {
    if (tl_params->algo == algo_t::ALGO_AUTO || tl_params->algo == algo_t::NAIVE) {
      int depth = max_depth(model);
      // max 2**25 dense nodes, 256 MiB dense model size. Categorical mask size is unlimited and not
      // affected by storage format.
      const int LOG2_MAX_DENSE_NODES = 25;
      int log2_num_dense_nodes       = depth + 1 + int(ceil(std::log2(model.trees.size())));
      storage_type = log2_num_dense_nodes > LOG2_MAX_DENSE_NODES ? storage_type_t::SPARSE
                                                                 : storage_type_t::DENSE;
    } else {
      // only dense storage is supported for other algorithms
      storage_type = storage_type_t::DENSE;
    }
  }

  forest_params_t params;
  cat_sets_owner cat_sets;
  switch (storage_type) {
    case storage_type_t::DENSE: {
      std::vector<dense_node> nodes;
      std::vector<float> vector_leaf;
      tl2fil_dense(&nodes, &params, model, tl_params, &cat_sets, &vector_leaf);
      init_dense(handle, pforest, cat_sets.accessor(), vector_leaf, nodes.data(), &params);
      // sync is necessary as nodes is used in init_dense(),
      // but destructed at the end of this function
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      if (tl_params->pforest_shape_str) {
        *tl_params->pforest_shape_str = sprintf_shape(model, storage_type, nodes, {}, cat_sets);
      }
      break;
    }
    case storage_type_t::SPARSE: {
      std::vector<int> trees;
      std::vector<sparse_node16> nodes;
      std::vector<float> vector_leaf;
      tl2fil_sparse(&trees, &nodes, &params, model, tl_params, &cat_sets, &vector_leaf);
      init_sparse(
        handle, pforest, cat_sets.accessor(), vector_leaf, trees.data(), nodes.data(), &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      if (tl_params->pforest_shape_str) {
        *tl_params->pforest_shape_str = sprintf_shape(model, storage_type, nodes, trees, cat_sets);
      }
      break;
    }
    case storage_type_t::SPARSE8: {
      std::vector<int> trees;
      std::vector<sparse_node8> nodes;
      std::vector<float> vector_leaf;
      tl2fil_sparse(&trees, &nodes, &params, model, tl_params, &cat_sets, &vector_leaf);
      init_sparse(
        handle, pforest, cat_sets.accessor(), vector_leaf, trees.data(), nodes.data(), &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      if (tl_params->pforest_shape_str) {
        *tl_params->pforest_shape_str = sprintf_shape(model, storage_type, nodes, trees, cat_sets);
      }
      break;
    }
    default: ASSERT(false, "tl_params->sparse must be one of AUTO, DENSE or SPARSE");
  }
}

void from_treelite(const raft::handle_t& handle,
                   forest_t* pforest,
                   ModelHandle model,
                   const treelite_params_t* tl_params)
{
  const tl::Model& model_ref = *(tl::Model*)model;
  model_ref.Dispatch([&](const auto& model_inner) {
    // model_inner is of the concrete type tl::ModelImpl<threshold_t, leaf_t>
    from_treelite(handle, pforest, model_inner, tl_params);
  });
}

// allocates caller-owned char* using malloc()
template <typename threshold_t, typename leaf_t, typename node_t>
char* sprintf_shape(const tl::ModelImpl<threshold_t, leaf_t>& model,
                    storage_type_t storage,
                    const std::vector<node_t>& nodes,
                    const std::vector<int>& trees,
                    const cat_sets_owner cat_sets)
{
  std::stringstream forest_shape = depth_hist_and_max(model);
  double size_mb = (trees.size() * sizeof(trees.front()) + nodes.size() * sizeof(nodes.front()) +
                    cat_sets.bits.size()) /
                   1e6;
  forest_shape << storage_type_repr[storage] << " model size " << std::setprecision(2) << size_mb
               << " MB" << std::endl;
  if (cat_sets.bits.size() > 0) {
    forest_shape << "number of categorical nodes for each feature id: {";
    std::size_t total_cat_nodes = 0;
    for (std::size_t n : cat_sets.n_nodes) {
      forest_shape << n << " ";
      total_cat_nodes += n;
    }
    forest_shape << "}" << std::endl << "total categorical nodes: " << total_cat_nodes << std::endl;
    forest_shape << "maximum matching category for each feature id: {";
    for (int mm : cat_sets.max_matching)
      forest_shape << mm << " ";
    forest_shape << "}" << std::endl;
  }
  // stream may be discontiguous
  std::string forest_shape_str = forest_shape.str();
  // now copy to a non-owning allocation
  char* shape_out = (char*)malloc(forest_shape_str.size() + 1);  // incl. \0
  memcpy((void*)shape_out, forest_shape_str.c_str(), forest_shape_str.size() + 1);
  return shape_out;
}

void free(const raft::handle_t& h, forest_t f)
{
  f->free(h);
  delete f;
}

void predict(const raft::handle_t& h,
             forest_t f,
             float* preds,
             const float* data,
             size_t num_rows,
             bool predict_proba)
{
  f->predict(h, preds, data, num_rows, predict_proba);
}

}  // namespace fil
}  // namespace ML
