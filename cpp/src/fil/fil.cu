#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file fil.cu fil.cu implements the forest data types (dense and sparse), including their
creation and prediction (the main inference kernel is defined in infer.cu). */

#include "common.cuh"    // for predict_params, sparse_storage, dense_storage
#include "internal.cuh"  // for cat_sets_device_owner, categorical_sets, output_t,

#include <cuml/fil/fil.h>  // for algo_t,

#include <raft/cudart_utils.h>     // for CUDA_CHECK, hipStream_t,
#include <thrust/host_vector.h>    // for host_vector
#include <raft/error.hpp>          // for ASSERT
#include <raft/handle.hpp>         // for handle_t
#include <rmm/device_uvector.hpp>  // for device_uvector

#include <cmath>    // for expf
#include <cstddef>  // for size_t

namespace ML {
namespace fil {

__host__ __device__ float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

/** performs additional transformations on the array of forest predictions
    (preds) of size n; the transformations are defined by output, and include
    averaging (multiplying by inv_num_trees), adding global_bias (always done),
    sigmoid and applying threshold. in case of complement_proba,
    fills in the complement probability */
__global__ void transform_k(float* preds,
                            size_t n,
                            output_t output,
                            float inv_num_trees,
                            float threshold,
                            float global_bias,
                            bool complement_proba)
{
  size_t i = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (i >= n) return;
  if (complement_proba && i % 2 != 0) return;

  float result = preds[i];
  if ((output & output_t::AVG) != 0) result *= inv_num_trees;
  result += global_bias;
  if ((output & output_t::SIGMOID) != 0) result = sigmoid(result);
  // will not be done on CATEGORICAL_LEAF because the whole kernel will not run
  if ((output & output_t::CLASS) != 0) { result = result > threshold ? 1.0f : 0.0f; }
  // sklearn outputs numpy array in 'C' order, with the number of classes being last dimension
  // that is also the default order, so we should use the same one
  if (complement_proba) {
    preds[i]     = 1.0f - result;
    preds[i + 1] = result;
  } else
    preds[i] = result;
}

// needed to avoid expanding the dispatch template into unresolved
// compute_smem_footprint::run() calls. In infer.cu, we don't export those symbols,
// but rather one symbol for the whole template specialization, as below.
extern template int dispatch_on_fil_template_params(compute_smem_footprint, predict_params);

struct forest {
  forest(const raft::handle_t& h) : vector_leaf_(0, h.get_stream()), cat_sets_(h.get_stream()) {}

  void init_n_items(int device)
  {
    int max_shm_std = 48 * 1024;  // 48 KiB
    /// the most shared memory a kernel can request on the GPU in question
    int max_shm = 0;
    CUDA_CHECK(hipDeviceGetAttribute(&max_shm, hipDeviceAttributeSharedMemPerBlockOptin, device));
    /* Our GPUs have been growing the shared memory size generation after
       generation. Eventually, a CUDA GPU might come by that supports more
       shared memory that would fit into unsigned 16-bit int. For such a GPU,
       we would have otherwise silently overflowed the index calculation due
       to short division. It would have failed cpp tests, but we might forget
       about this source of bugs, if not for the failing assert. */
    ASSERT(max_shm < 262144,
           "internal error: please use a larger type inside"
           " infer_k for column count");
    // TODO(canonizer): use >48KiB shared memory if available
    max_shm = std::min(max_shm, max_shm_std);

    // searching for the most items per block while respecting the shared
    // memory limits creates a full linear programming problem.
    // solving it in a single equation looks less tractable than this
    for (bool predict_proba : {false, true}) {
      shmem_size_params& ssp_ = predict_proba ? proba_ssp_ : class_ssp_;
      ssp_.predict_proba      = predict_proba;
      shmem_size_params ssp   = ssp_;
      // if n_items was not provided, try from 1 to MAX_N_ITEMS. Otherwise, use as-is.
      int min_n_items = ssp.n_items == 0 ? 1 : ssp.n_items;
      int max_n_items =
        ssp.n_items == 0 ? (algo_ == algo_t::BATCH_TREE_REORG ? MAX_N_ITEMS : 1) : ssp.n_items;
      for (bool cols_in_shmem : {false, true}) {
        ssp.cols_in_shmem = cols_in_shmem;
        for (ssp.n_items = min_n_items; ssp.n_items <= max_n_items; ++ssp.n_items) {
          ssp.shm_sz = dispatch_on_fil_template_params(compute_smem_footprint(), ssp);
          if (ssp.shm_sz < max_shm) ssp_ = ssp;
        }
      }
      ASSERT(max_shm >= ssp_.shm_sz,
             "FIL out of shared memory. Perhaps the maximum number of \n"
             "supported classes is exceeded? 5'000 would still be safe.");
    }
  }

  void init_fixed_block_count(int device, int blocks_per_sm)
  {
    int max_threads_per_sm, sm_count;
    CUDA_CHECK(
      hipDeviceGetAttribute(&max_threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, device));
    blocks_per_sm = std::min(blocks_per_sm, max_threads_per_sm / FIL_TPB);
    CUDA_CHECK(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device));
    fixed_block_count_ = blocks_per_sm * sm_count;
  }

  void init_common(const raft::handle_t& h,
                   const categorical_sets& cat_sets,
                   const std::vector<float>& vector_leaf,
                   const forest_params_t* params)
  {
    depth_                           = params->depth;
    num_trees_                       = params->num_trees;
    algo_                            = params->algo;
    output_                          = params->output;
    threshold_                       = params->threshold;
    global_bias_                     = params->global_bias;
    proba_ssp_.n_items               = params->n_items;
    proba_ssp_.log2_threads_per_tree = log2(params->threads_per_tree);
    proba_ssp_.leaf_algo             = params->leaf_algo;
    proba_ssp_.num_cols              = params->num_cols;
    proba_ssp_.num_classes           = params->num_classes;
    proba_ssp_.cats_present          = cat_sets.cats_present();
    class_ssp_                       = proba_ssp_;

    int device          = h.get_device();
    hipStream_t stream = h.get_stream();
    init_n_items(device);  // n_items takes priority over blocks_per_sm
    init_fixed_block_count(device, params->blocks_per_sm);

    // vector leaf
    if (!vector_leaf.empty()) {
      vector_leaf_.resize(vector_leaf.size(), stream);

      CUDA_CHECK(hipMemcpyAsync(vector_leaf_.data(),
                                 vector_leaf.data(),
                                 vector_leaf.size() * sizeof(float),
                                 hipMemcpyHostToDevice,
                                 stream));
    }

    // categorical features
    cat_sets_ = cat_sets_device_owner(cat_sets, stream);
  }

  virtual void infer(predict_params params, hipStream_t stream) = 0;

  void predict(
    const raft::handle_t& h, float* preds, const float* data, size_t num_rows, bool predict_proba)
  {
    // Initialize prediction parameters.
    predict_params params(predict_proba ? proba_ssp_ : class_ssp_);
    params.algo     = algo_;
    params.preds    = preds;
    params.data     = data;
    params.num_rows = num_rows;
    // ignored unless predict_proba is true and algo is GROVE_PER_CLASS
    params.transform = output_;
    // fixed_block_count_ == 0 means the number of thread blocks is
    // proportional to the number of rows
    params.num_blocks = fixed_block_count_;

    /**
    The binary classification / regression (FLOAT_UNARY_BINARY) predict_proba() works as follows
      (always 2 outputs):
    RAW: output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error
    write the output of the previous stages and its complement

    The binary classification / regression (FLOAT_UNARY_BINARY) predict() works as follows
      (always 1 output):
    RAW (no values set): output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: apply threshold (equivalent to choosing best class)
    SOFTMAX is set: error

    The multi-class classification / regression (CATEGORICAL_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error

    The multi-class classification / regression (CATEGORICAL_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest probability, else output label
    0. SOFTMAX is set: error All other flags (AVG, SIGMOID, CLASS) are ignored

    The multi-class classification / regression (GROVE_PER_CLASS) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid; if SOFTMAX is also set: error
    CLASS is set: ignored
    SOFTMAX is set: softmax is applied after averaging and global_bias

    The multi-class classification / regression (GROVE_PER_CLASS) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS, SOFTMAX) are ignored

    The multi-class classification / regression (VECTOR_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid; if SOFTMAX is also set: error
    CLASS is set: ignored
    SOFTMAX is set: softmax is applied after averaging and global_bias
    All other flags (SIGMOID, CLASS, SOFTMAX) are ignored

    The multi-class classification / regression (VECTOR_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS, SOFTMAX) are ignored
    */
    output_t ot = output_;
    // Treelite applies bias before softmax, but we do after.
    // Simulating treelite order, which cancels out bias.
    // If non-proba prediction used, it still will not matter
    // for the same reason softmax will not.
    float global_bias     = (ot & output_t::SOFTMAX) != 0 ? 0.0f : global_bias_;
    bool complement_proba = false, do_transform;

    if (predict_proba) {
      // no threshold on probabilities
      ot = output_t(ot & ~output_t::CLASS);

      switch (params.leaf_algo) {
        case leaf_algo_t::FLOAT_UNARY_BINARY:
          params.num_outputs = 2;
          complement_proba   = true;
          do_transform       = true;
          break;
        case leaf_algo_t::GROVE_PER_CLASS:
          // for GROVE_PER_CLASS, averaging happens in infer_k
          ot                 = output_t(ot & ~output_t::AVG);
          params.num_outputs = params.num_classes;
          do_transform = (ot != output_t::RAW && ot != output_t::SOFTMAX) || global_bias != 0.0f;
          break;
        case leaf_algo_t::CATEGORICAL_LEAF:
          params.num_outputs = params.num_classes;
          do_transform       = ot != output_t::RAW || global_bias_ != 0.0f;
          break;
        case leaf_algo_t::VECTOR_LEAF:
          // for VECTOR_LEAF, averaging happens in infer_k
          ot                 = output_t(ot & ~output_t::AVG);
          params.num_outputs = params.num_classes;
          do_transform = (ot != output_t::RAW && ot != output_t::SOFTMAX) || global_bias != 0.0f;
          break;
        default: ASSERT(false, "internal error: predict: invalid leaf_algo %d", params.leaf_algo);
      }
    } else {
      if (params.leaf_algo == leaf_algo_t::FLOAT_UNARY_BINARY) {
        do_transform = ot != output_t::RAW || global_bias_ != 0.0f;
      } else {
        // GROVE_PER_CLASS, CATEGORICAL_LEAF: moot since choosing best class and
        // all transforms are monotonic. also, would break current code
        do_transform = false;
      }
      params.num_outputs = 1;
    }

    // Predict using the forest.
    hipStream_t stream = h.get_stream();
    infer(params, stream);

    if (do_transform) {
      size_t num_values_to_transform = (size_t)num_rows * (size_t)params.num_outputs;
      transform_k<<<raft::ceildiv(num_values_to_transform, (size_t)FIL_TPB), FIL_TPB, 0, stream>>>(
        preds,
        num_values_to_transform,
        ot,
        num_trees_ > 0 ? (1.0f / num_trees_) : 1.0f,
        threshold_,
        global_bias,
        complement_proba);
      CUDA_CHECK(hipPeekAtLastError());
    }
  }

  virtual void free(const raft::handle_t& h)
  {
    cat_sets_.release();
    vector_leaf_.release();
  }

  virtual ~forest() {}

  int num_trees_     = 0;
  int depth_         = 0;
  algo_t algo_       = algo_t::NAIVE;
  output_t output_   = output_t::RAW;
  float threshold_   = 0.5;
  float global_bias_ = 0;
  shmem_size_params class_ssp_, proba_ssp_;
  int fixed_block_count_ = 0;
  // Optionally used
  rmm::device_uvector<float> vector_leaf_;
  cat_sets_device_owner cat_sets_;
};

struct dense_forest : forest {
  dense_forest(const raft::handle_t& h) : forest(h), nodes_(0, h.get_stream()) {}

  void transform_trees(const dense_node* nodes)
  {
    /* Populate node information:
       For each tree, the nodes are still stored in the breadth-first,
       left-to-right order. However, instead of storing the nodes of the same
       tree adjacently, it uses a different layout. In this layout, the roots
       of all trees (node 0) are stored first, followed by left children of
       the roots of all trees (node 1), followed by the right children of the
       roots of all trees (node 2), and so on.
    */
    int global_node = 0;
    for (int tree = 0; tree < num_trees_; ++tree) {
      int tree_node = 0;
      // the counters `level` and `branch` are not used for computing node
      // indices, they are only here to highlight the node ordering within
      // each tree
      for (int level = 0; level <= depth_; ++level) {
        for (int branch = 0; branch < 1 << level; ++branch) {
          h_nodes_[tree_node * num_trees_ + tree] = nodes[global_node];
          ++tree_node;
          ++global_node;
        }
      }
    }
  }

  void init(const raft::handle_t& h,
            const categorical_sets& cat_sets,
            const std::vector<float>& vector_leaf,
            const dense_node* nodes,
            const forest_params_t* params)
  {
    init_common(h, cat_sets, vector_leaf, params);
    if (algo_ == algo_t::NAIVE) algo_ = algo_t::BATCH_TREE_REORG;

    int num_nodes = forest_num_nodes(num_trees_, depth_);
    nodes_.resize(num_nodes, h.get_stream());
    h_nodes_.resize(num_nodes);
    if (algo_ == algo_t::NAIVE) {
      std::copy(nodes, nodes + num_nodes, h_nodes_.begin());
    } else {
      transform_trees(nodes);
    }
    CUDA_CHECK(hipMemcpyAsync(nodes_.data(),
                               h_nodes_.data(),
                               num_nodes * sizeof(dense_node),
                               hipMemcpyHostToDevice,
                               h.get_stream()));
    // copy must be finished before freeing the host data
    CUDA_CHECK(hipStreamSynchronize(h.get_stream()));
    h_nodes_.clear();
    h_nodes_.shrink_to_fit();
  }

  virtual void infer(predict_params params, hipStream_t stream) override
  {
    dense_storage forest(cat_sets_.accessor(),
                         vector_leaf_.data(),
                         nodes_.data(),
                         num_trees_,
                         algo_ == algo_t::NAIVE ? tree_num_nodes(depth_) : 1,
                         algo_ == algo_t::NAIVE ? 1 : num_trees_);
    fil::infer(forest, params, stream);
  }

  virtual void free(const raft::handle_t& h) override
  {
    nodes_.release();
    forest::free(h);
  }

  rmm::device_uvector<dense_node> nodes_;
  thrust::host_vector<dense_node> h_nodes_;
};

template <typename node_t>
struct sparse_forest : forest {
  sparse_forest(const raft::handle_t& h)
    : forest(h), trees_(0, h.get_stream()), nodes_(0, h.get_stream())
  {
  }

  void init(const raft::handle_t& h,
            const categorical_sets& cat_sets,
            const std::vector<float>& vector_leaf,
            const int* trees,
            const node_t* nodes,
            const forest_params_t* params)
  {
    init_common(h, cat_sets, vector_leaf, params);
    if (algo_ == algo_t::ALGO_AUTO) algo_ = algo_t::NAIVE;
    depth_     = 0;  // a placeholder value
    num_nodes_ = params->num_nodes;

    // trees
    trees_.resize(num_trees_, h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(
      trees_.data(), trees, sizeof(int) * num_trees_, hipMemcpyHostToDevice, h.get_stream()));

    // nodes
    nodes_.resize(num_nodes_, h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(
      nodes_.data(), nodes, sizeof(node_t) * num_nodes_, hipMemcpyHostToDevice, h.get_stream()));
  }

  virtual void infer(predict_params params, hipStream_t stream) override
  {
    sparse_storage<node_t> forest(
      cat_sets_.accessor(), vector_leaf_.data(), trees_.data(), nodes_.data(), num_trees_);
    fil::infer(forest, params, stream);
  }

  void free(const raft::handle_t& h) override
  {
    forest::free(h);
    trees_.release();
    nodes_.release();
  }

  int num_nodes_ = 0;
  rmm::device_uvector<int> trees_;
  rmm::device_uvector<node_t> nodes_;
};

void check_params(const forest_params_t* params, bool dense)
{
  if (dense) {
    ASSERT(params->depth >= 0, "depth must be non-negative for dense forests");
  } else {
    ASSERT(params->num_nodes >= 0, "num_nodes must be non-negative for sparse forests");
    ASSERT(params->algo == algo_t::NAIVE || params->algo == algo_t::ALGO_AUTO,
           "only ALGO_AUTO and NAIVE algorithms are supported "
           "for sparse forests");
  }
  ASSERT(params->num_trees >= 0, "num_trees must be non-negative");
  ASSERT(params->num_cols >= 0, "num_cols must be non-negative");
  switch (params->algo) {
    case algo_t::ALGO_AUTO:
    case algo_t::NAIVE:
    case algo_t::TREE_REORG:
    case algo_t::BATCH_TREE_REORG: break;
    default: ASSERT(false, "algo should be ALGO_AUTO, NAIVE, TREE_REORG or BATCH_TREE_REORG");
  }
  switch (params->leaf_algo) {
    case leaf_algo_t::FLOAT_UNARY_BINARY:
      if ((params->output & output_t::CLASS) != 0) {
        ASSERT(params->num_classes == 2,
               "only supporting binary"
               " classification using FLOAT_UNARY_BINARY");
      } else {
        ASSERT(params->num_classes == 1,
               "num_classes must be 1 for "
               "regression");
      }
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax does not make sense for leaf_algo == FLOAT_UNARY_BINARY");
      break;
    case leaf_algo_t::GROVE_PER_CLASS:
      ASSERT(params->threads_per_tree == 1, "multiclass not supported with threads_per_tree > 1");
      ASSERT(params->num_classes > 2,
             "num_classes > 2 is required for leaf_algo == GROVE_PER_CLASS");
      ASSERT(params->num_trees % params->num_classes == 0,
             "num_classes must divide num_trees evenly for GROVE_PER_CLASS");
      break;
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(params->threads_per_tree == 1, "multiclass not supported with threads_per_tree > 1");
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == CATEGORICAL_LEAF");
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax not supported for leaf_algo == CATEGORICAL_LEAF");
      break;
    case leaf_algo_t::VECTOR_LEAF:
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == VECTOR_LEAF");
      break;
    default:
      ASSERT(false,
             "leaf_algo must be FLOAT_UNARY_BINARY, CATEGORICAL_LEAF"
             " or GROVE_PER_CLASS");
  }
  // output_t::RAW == 0, and doesn't have a separate flag
  if ((params->output & ~output_t::ALL_SET) != 0) {
    ASSERT(false, "output should be a combination of RAW, AVG, SIGMOID, CLASS and SOFTMAX");
  }
  ASSERT(~params->output & (output_t::SIGMOID | output_t::SOFTMAX),
         "combining softmax and sigmoid is not supported");
  ASSERT(params->blocks_per_sm >= 0, "blocks_per_sm must be nonnegative");
  ASSERT(params->n_items >= 0, "n_items must be non-negative");
  ASSERT(params->threads_per_tree > 0, "threads_per_tree must be positive");
  ASSERT(thrust::detail::is_power_of_2(params->threads_per_tree),
         "threads_per_tree must be a power of 2");
  ASSERT(params->threads_per_tree <= FIL_TPB,
         "threads_per_tree must not "
         "exceed block size %d",
         FIL_TPB);
}

void init_dense(const raft::handle_t& h,
                forest_t* pf,
                const categorical_sets& cat_sets,
                const std::vector<float>& vector_leaf,
                const dense_node* nodes,
                const forest_params_t* params)
{
  check_params(params, true);
  dense_forest* f = new dense_forest(h);
  f->init(h, cat_sets, vector_leaf, nodes, params);
  *pf = f;
}

template <typename fil_node_t>
void init_sparse(const raft::handle_t& h,
                 forest_t* pf,
                 const categorical_sets& cat_sets,
                 const std::vector<float>& vector_leaf,
                 const int* trees,
                 const fil_node_t* nodes,
                 const forest_params_t* params)
{
  check_params(params, false);
  sparse_forest<fil_node_t>* f = new sparse_forest<fil_node_t>(h);
  f->init(h, cat_sets, vector_leaf, trees, nodes, params);
  *pf = f;
}

// explicit instantiations for init_sparse()
template void init_sparse<sparse_node16>(const raft::handle_t& h,
                                         forest_t* pf,
                                         const categorical_sets& cat_sets,
                                         const std::vector<float>& vector_leaf,
                                         const int* trees,
                                         const sparse_node16* nodes,
                                         const forest_params_t* params);

template void init_sparse<sparse_node8>(const raft::handle_t& h,
                                        forest_t* pf,
                                        const categorical_sets& cat_sets,
                                        const std::vector<float>& vector_leaf,
                                        const int* trees,
                                        const sparse_node8* nodes,
                                        const forest_params_t* params);

void free(const raft::handle_t& h, forest_t f)
{
  f->free(h);
  delete f;
}

void predict(const raft::handle_t& h,
             forest_t f,
             float* preds,
             const float* data,
             size_t num_rows,
             bool predict_proba)
{
  f->predict(h, preds, data, num_rows, predict_proba);
}

}  // namespace fil
}  // namespace ML
