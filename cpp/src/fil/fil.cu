#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file fil.cu implements forest inference */

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <treelite/c_api.h>
#include <treelite/tree.h>
#include <algorithm>
#include <cmath>
#include <limits>
#include <stack>
#include <utility>

#include <cuml/fil/fil.h>
#include <raft/cudart_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;
namespace tl = treelite;

void node_init(dense_node_t* n, val_t output, float thresh, int fid,
               bool def_left, bool is_leaf) {
  *n = dense_node(output, thresh, fid, def_left, is_leaf);
}

void node_decode(const dense_node_t* n, val_t* output, float* thresh, int* fid,
                 bool* def_left, bool* is_leaf) {
  dense_node dn(*n);
  *output = dn.output<val_t>();
  *thresh = dn.thresh();
  *fid = dn.fid();
  *def_left = dn.def_left();
  *is_leaf = dn.is_leaf();
}

inline void node_init_inline(sparse_node16_t* node, val_t output, float thresh,
                             int fid, bool def_left, bool is_leaf,
                             int left_index) {
  sparse_node16 n(output, thresh, fid, def_left, is_leaf, left_index);
  *node = sparse_node16_t(n, n);
}

void node_init(sparse_node16_t* node, val_t output, float thresh, int fid,
               bool def_left, bool is_leaf, int left_index) {
  node_init_inline(node, output, thresh, fid, def_left, is_leaf, left_index);
}

void node_decode(const sparse_node16_t* node, val_t* output, float* thresh,
                 int* fid, bool* def_left, bool* is_leaf, int* left_index) {
  node_decode((const dense_node_t*)node, output, thresh, fid, def_left,
              is_leaf);
  *left_index = sparse_node16(*node).left_index();
}

inline void node_init_inline(sparse_node8_t* node, val_t output, float thresh,
                             int fid, bool def_left, bool is_leaf,
                             int left_index) {
  sparse_node8 n(output, thresh, fid, def_left, is_leaf, left_index);
  *node = sparse_node8_t(n);
}

void node_init(sparse_node8_t* node, val_t output, float thresh, int fid,
               bool def_left, bool is_leaf, int left_index) {
  node_init_inline(node, output, thresh, fid, def_left, is_leaf, left_index);
}

void node_decode(const sparse_node8_t* node, val_t* output, float* thresh,
                 int* fid, bool* def_left, bool* is_leaf, int* left_index) {
  node_decode((const dense_node_t*)node, output, thresh, fid, def_left,
              is_leaf);
  *left_index = sparse_node8(*node).left_index();
}

__host__ __device__ float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

/** performs additional transformations on the array of forest predictions
    (preds) of size n; the transformations are defined by output, and include
    averaging (multiplying by inv_num_trees), adding global_bias (always done),
    sigmoid and applying threshold. in case of complement_proba,
    fills in the complement probability */
__global__ void transform_k(float* preds, size_t n, output_t output,
                            float inv_num_trees, float threshold,
                            float global_bias, bool complement_proba) {
  size_t i = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (i >= n) return;
  if (complement_proba && i % 2 != 0) return;

  float result = preds[i];
  if ((output & output_t::AVG) != 0) result *= inv_num_trees;
  result += global_bias;
  if ((output & output_t::SIGMOID) != 0) result = sigmoid(result);
  // will not be done on CATEGORICAL_LEAF because the whole kernel will not run
  if ((output & output_t::CLASS) != 0) {
    result = result > threshold ? 1.0f : 0.0f;
  }
  // sklearn outputs numpy array in 'C' order, with the number of classes being last dimension
  // that is also the default order, so we should use the same one
  if (complement_proba) {
    preds[i] = 1.0f - result;
    preds[i + 1] = result;
  } else
    preds[i] = result;
}

struct forest {
  void init_max_shm() {
    int max_shm_std = 48 * 1024;  // 48 KiB
    int device = 0;
    // TODO(canonizer): use raft::handle_t for this
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipDeviceGetAttribute(
      &max_shm_, hipDeviceAttributeSharedMemPerBlockOptin, device));
    // TODO(canonizer): use >48KiB shared memory if available
    max_shm_ = std::min(max_shm_, max_shm_std);
  }

  void init_fixed_block_count(const raft::handle_t& h, int blocks_per_sm) {
    int max_threads_per_sm, sm_count;
    CUDA_CHECK(hipDeviceGetAttribute(&max_threads_per_sm,
                                      hipDeviceAttributeMaxThreadsPerMultiProcessor,
                                      h.get_device()));
    int max_blocks_per_sm = max_threads_per_sm / FIL_TPB;
    ASSERT(blocks_per_sm <= max_blocks_per_sm,
           "on this GPU, FIL blocks_per_sm cannot exceed %d",
           max_blocks_per_sm);
    CUDA_CHECK(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount,
                                      h.get_device()));
    fixed_block_count_ = blocks_per_sm * sm_count;
  }

  void init_common(const raft::handle_t& h, const forest_params_t* params) {
    depth_ = params->depth;
    num_trees_ = params->num_trees;
    num_cols_ = params->num_cols;
    algo_ = params->algo;
    output_ = params->output;
    threshold_ = params->threshold;
    global_bias_ = params->global_bias;
    leaf_algo_ = params->leaf_algo;
    num_classes_ = params->num_classes;
    init_max_shm();
    init_fixed_block_count(h, params->blocks_per_sm);
  }

  virtual void infer(predict_params params, hipStream_t stream) = 0;

  void predict(const raft::handle_t& h, float* preds, const float* data,
               size_t num_rows, bool predict_proba) {
    // Initialize prediction parameters.
    predict_params params;
    params.num_cols = num_cols_;
    params.algo = algo_;
    params.preds = preds;
    params.data = data;
    params.num_rows = num_rows;
    params.max_shm = max_shm_;
    params.num_classes = num_classes_;
    params.leaf_algo = leaf_algo_;
    // fixed_block_count_ == 0 means the number of thread blocks is
    // proportional to the number of rows
    params.num_blocks = fixed_block_count_;

    /**
    The binary classification / regression (FLOAT_UNARY_BINARY) predict_proba() works as follows
      (always 2 outputs):
    RAW: output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    write the output of the previous stages and its complement

    The binary classification / regression (FLOAT_UNARY_BINARY) predict() works as follows
      (always 1 output):
    RAW (no values set): output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: apply threshold (equivalent to choosing best class)
    
    The multi-class classification / regression (CATEGORICAL_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    
    The multi-class classification / regression (CATEGORICAL_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest probability, else output label 0.
    All other flags (AVG, SIGMOID, CLASS) are ignored
    
    The multi-class classification / regression (GROVE_PER_CLASS) predict_proba() is not implemented
    
    The multi-class classification / regression (GROVE_PER_CLASS) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS) are ignored
    */
    output_t ot = output_;
    bool complement_proba = false, do_transform;

    if (predict_proba) {
      // no threshold on probabilities
      ot = output_t(ot & ~output_t::CLASS);

      switch (leaf_algo_) {
        case leaf_algo_t::FLOAT_UNARY_BINARY:
          params.num_outputs = 2;
          complement_proba = true;
          do_transform = true;
          break;
        case leaf_algo_t::GROVE_PER_CLASS:
          // TODO(levsnv): add softmax to implement predict_proba
          ASSERT(
            false,
            "predict_proba not supported for multi-class gradient boosted "
            "decision trees (encountered in xgboost, scikit-learn, lightgbm)");
        case leaf_algo_t::CATEGORICAL_LEAF:
          params.num_outputs = num_classes_;
          do_transform = ot != output_t::RAW || global_bias_ != 0.0f;
          break;
        default:
          ASSERT(false, "internal error: invalid leaf_algo_");
      }
    } else {
      if (leaf_algo_ == leaf_algo_t::FLOAT_UNARY_BINARY) {
        do_transform = ot != output_t::RAW || global_bias_ != 0.0f;
      } else {
        // GROVE_PER_CLASS, CATEGORICAL_LEAF: moot since choosing best class and
        // all transforms are monotonic. also, would break current code
        do_transform = false;
      }
      params.num_outputs = 1;
    }

    // Predict using the forest.
    hipStream_t stream = h.get_stream();
    infer(params, stream);

    if (do_transform) {
      size_t num_values_to_transform =
        (size_t)num_rows * (size_t)params.num_outputs;
      transform_k<<<raft::ceildiv(num_values_to_transform, (size_t)FIL_TPB),
                    FIL_TPB, 0, stream>>>(
        preds, num_values_to_transform, ot,
        num_trees_ > 0 ? (1.0f / num_trees_) : 1.0f, threshold_, global_bias_,
        complement_proba);
      CUDA_CHECK(hipPeekAtLastError());
    }
  }

  virtual void free(const raft::handle_t& h) = 0;
  virtual ~forest() {}

  int num_trees_ = 0;
  int depth_ = 0;
  int num_cols_ = 0;
  algo_t algo_ = algo_t::NAIVE;
  int max_shm_ = 0;
  output_t output_ = output_t::RAW;
  float threshold_ = 0.5;
  float global_bias_ = 0;
  leaf_algo_t leaf_algo_ = leaf_algo_t::FLOAT_UNARY_BINARY;
  int num_classes_ = 1;
  int fixed_block_count_ = 0;
};

struct dense_forest : forest {
  void transform_trees(const dense_node_t* nodes) {
    /* Populate node information:
       For each tree, the nodes are still stored in the breadth-first,
       left-to-right order. However, instead of storing the nodes of the same
       tree adjacently, it uses a different layout. In this layout, the roots
       of all trees (node 0) are stored first, followed by left children of
       the roots of all trees (node 1), followed by the right children of the
       roots of all trees (node 2), and so on.
    */
    int global_node = 0;
    for (int tree = 0; tree < num_trees_; ++tree) {
      int tree_node = 0;
      // the counters `level` and `branch` are not used for computing node
      // indices, they are only here to highlight the node ordering within
      // each tree
      for (int level = 0; level <= depth_; ++level) {
        for (int branch = 0; branch < 1 << level; ++branch) {
          h_nodes_[tree_node * num_trees_ + tree] =
            dense_node(nodes[global_node]);
          ++tree_node;
          ++global_node;
        }
      }
    }
  }

  void init(const raft::handle_t& h, const dense_node_t* nodes,
            const forest_params_t* params) {
    init_common(h, params);
    if (algo_ == algo_t::NAIVE) algo_ = algo_t::BATCH_TREE_REORG;

    int num_nodes = forest_num_nodes(num_trees_, depth_);
    nodes_ = (dense_node*)h.get_device_allocator()->allocate(
      sizeof(dense_node) * num_nodes, h.get_stream());
    h_nodes_.resize(num_nodes);
    if (algo_ == algo_t::NAIVE) {
      std::copy(nodes, nodes + num_nodes, h_nodes_.begin());
    } else {
      transform_trees(nodes);
    }
    CUDA_CHECK(hipMemcpyAsync(nodes_, h_nodes_.data(),
                               num_nodes * sizeof(dense_node),
                               hipMemcpyHostToDevice, h.get_stream()));
    // copy must be finished before freeing the host data
    CUDA_CHECK(hipStreamSynchronize(h.get_stream()));
    h_nodes_.clear();
    h_nodes_.shrink_to_fit();
  }

  virtual void infer(predict_params params, hipStream_t stream) override {
    dense_storage forest(nodes_, num_trees_,
                         algo_ == algo_t::NAIVE ? tree_num_nodes(depth_) : 1,
                         algo_ == algo_t::NAIVE ? 1 : num_trees_);
    fil::infer(forest, params, stream);
  }

  virtual void free(const raft::handle_t& h) override {
    int num_nodes = forest_num_nodes(num_trees_, depth_);
    h.get_device_allocator()->deallocate(nodes_, sizeof(dense_node) * num_nodes,
                                         h.get_stream());
  }

  dense_node* nodes_ = nullptr;
  thrust::host_vector<dense_node> h_nodes_;
};

template <typename node_t>
struct external_node {};

template <>
struct external_node<sparse_node16> {
  typedef sparse_node16_t t;
};

template <>
struct external_node<sparse_node8> {
  typedef sparse_node8_t t;
};

template <typename node_t>
struct sparse_forest : forest {
  typedef typename external_node<node_t>::t external_node_t;
  void init(const raft::handle_t& h, const int* trees,
            const external_node_t* nodes, const forest_params_t* params) {
    init_common(h, params);
    if (algo_ == algo_t::ALGO_AUTO) algo_ = algo_t::NAIVE;
    depth_ = 0;  // a placeholder value
    num_nodes_ = params->num_nodes;

    // trees
    trees_ = (int*)h.get_device_allocator()->allocate(sizeof(int) * num_trees_,
                                                      h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(trees_, trees, sizeof(int) * num_trees_,
                               hipMemcpyHostToDevice, h.get_stream()));

    // nodes
    nodes_ = (node_t*)h.get_device_allocator()->allocate(
      sizeof(node_t) * num_nodes_, h.get_stream());
    CUDA_CHECK(hipMemcpyAsync(nodes_, nodes, sizeof(node_t) * num_nodes_,
                               hipMemcpyHostToDevice, h.get_stream()));
  }

  virtual void infer(predict_params params, hipStream_t stream) override {
    sparse_storage<node_t> forest(trees_, nodes_, num_trees_);
    fil::infer(forest, params, stream);
  }

  void free(const raft::handle_t& h) override {
    h.get_device_allocator()->deallocate(trees_, sizeof(int) * num_trees_,
                                         h.get_stream());
    h.get_device_allocator()->deallocate(nodes_, sizeof(node_t) * num_nodes_,
                                         h.get_stream());
  }

  int num_nodes_ = 0;
  int* trees_ = nullptr;
  node_t* nodes_ = nullptr;
};

void check_params(const forest_params_t* params, bool dense) {
  if (dense) {
    ASSERT(params->depth >= 0, "depth must be non-negative for dense forests");
  } else {
    ASSERT(params->num_nodes >= 0,
           "num_nodes must be non-negative for sparse forests");
    ASSERT(params->algo == algo_t::NAIVE || params->algo == algo_t::ALGO_AUTO,
           "only ALGO_AUTO and NAIVE algorithms are supported "
           "for sparse forests");
  }
  ASSERT(params->num_trees >= 0, "num_trees must be non-negative");
  ASSERT(params->num_cols >= 0, "num_cols must be non-negative");
  switch (params->algo) {
    case algo_t::ALGO_AUTO:
    case algo_t::NAIVE:
    case algo_t::TREE_REORG:
    case algo_t::BATCH_TREE_REORG:
      break;
    default:
      ASSERT(false,
             "algo should be ALGO_AUTO, NAIVE, TREE_REORG or BATCH_TREE_REORG");
  }
  switch (params->leaf_algo) {
    case leaf_algo_t::FLOAT_UNARY_BINARY:
      if ((params->output & output_t::CLASS) != 0) {
        ASSERT(params->num_classes == 2,
               "only supporting binary"
               " classification using FLOAT_UNARY_BINARY");
      } else {
        ASSERT(params->num_classes == 1,
               "num_classes must be 1 for "
               "regression");
      }
      break;
    case leaf_algo_t::GROVE_PER_CLASS:
      ASSERT(params->num_classes > 2,
             "num_classes > 2 is required for leaf_algo == GROVE_PER_CLASS");
      ASSERT(params->num_trees % params->num_classes == 0,
             "num_classes must divide num_trees evenly for GROVE_PER_CLASS");
      break;
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == CATEGORICAL_LEAF");
      break;
    default:
      ASSERT(false,
             "leaf_algo must be FLOAT_UNARY_BINARY, CATEGORICAL_LEAF"
             " or GROVE_PER_CLASS");
  }
  // output_t::RAW == 0, and doesn't have a separate flag
  output_t all_set =
    output_t(output_t::AVG | output_t::SIGMOID | output_t::CLASS);
  if ((params->output & ~all_set) != 0) {
    ASSERT(false,
           "output should be a combination of RAW, AVG, SIGMOID and CLASS");
  }
  ASSERT(params->blocks_per_sm >= 0, "blocks_per_sm must be nonnegative");
}

int tree_root(const tl::Tree& tree) {
  return 0;  // Treelite format assumes that the root is 0
}

int max_depth_helper(const tl::Tree& tree, int node_id, int limit) {
  if (tree.IsLeaf(node_id)) return 0;
  ASSERT(limit > 0,
         "recursion depth limit reached, might be a cycle in the tree");
  return 1 +
         std::max(max_depth_helper(tree, tree.LeftChild(node_id), limit - 1),
                  max_depth_helper(tree, tree.RightChild(node_id), limit - 1));
}

inline int max_depth(const tl::Tree& tree) {
  // trees of this depth aren't used, so it most likely means bad input data,
  // e.g. cycles in the forest
  const int DEPTH_LIMIT = 500;
  int root_index = tree_root(tree);
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  stack.push(pair_t(root_index, 0));
  int max_depth = 0;
  while (!stack.empty()) {
    const pair_t& pair = stack.top();
    int node_id = pair.first;
    int depth = pair.second;
    stack.pop();
    while (!tree.IsLeaf(node_id)) {
      stack.push(pair_t(tree.LeftChild(node_id), depth + 1));
      node_id = tree.RightChild(node_id);
      depth++;
      ASSERT(depth < DEPTH_LIMIT,
             "depth limit reached, might be a cycle in the tree");
    }
    // only need to update depth for leaves
    max_depth = std::max(max_depth, depth);
  }
  return max_depth;
}

int max_depth(const tl::Model& model) {
  int depth = 0;
  for (const auto& tree : model.trees) depth = std::max(depth, max_depth(tree));
  return depth;
}

inline void adjust_threshold(float* pthreshold, int* tl_left, int* tl_right,
                             bool* default_left, tl::Operator comparison_op) {
  // in treelite (take left node if val [op] threshold),
  // the meaning of the condition is reversed compared to FIL;
  // thus, "<" in treelite corresonds to comparison ">=" used by FIL
  // https://github.com/dmlc/treelite/blob/master/include/treelite/tree.h#L243
  switch (comparison_op) {
    case tl::Operator::kLT:
      break;
    case tl::Operator::kLE:
      // x <= y is equivalent to x < y', where y' is the next representable float
      *pthreshold =
        std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
      break;
    case tl::Operator::kGT:
      // x > y is equivalent to x >= y', where y' is the next representable float
      // left and right still need to be swapped
      *pthreshold =
        std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
    case tl::Operator::kGE:
      // swap left and right
      std::swap(*tl_left, *tl_right);
      *default_left = !*default_left;
      break;
    default:
      ASSERT(false, "only <, >, <= and >= comparisons are supported");
  }
}

/** if the vector consists of zeros and a single one, return the position
for the one (assumed class label). Else, asserts false.
If the vector contains a NAN, asserts false */
int find_class_label_from_one_hot(tl::tl_float* vector, int len) {
  bool found_label = false;
  int out;
  for (int i = 0; i < len; ++i) {
    if (vector[i] == 1.0f) {
      ASSERT(!found_label, "label vector contains multiple 1.0f");
      out = i;
      found_label = true;
    } else {
      ASSERT(vector[i] == 0.0f,
             "label vector contains values other than 0.0 and 1.0");
    }
  }
  ASSERT(found_label, "did not find 1.0f in vector");
  return out;
}

template <typename fil_node_t>
void tl2fil_leaf_payload(fil_node_t* fil_node, const tl::Tree& tl_tree,
                         int tl_node_id, const forest_params_t& forest_params) {
  auto vec = tl_tree.LeafVector(tl_node_id);
  switch (forest_params.leaf_algo) {
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(vec.size() == forest_params.num_classes,
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = find_class_label_from_one_hot(&vec[0], vec.size());
      break;
    case leaf_algo_t::FLOAT_UNARY_BINARY:
    case leaf_algo_t::GROVE_PER_CLASS:
      fil_node->val.f = tl_tree.LeafValue(tl_node_id);
      ASSERT(!tl_tree.HasLeafVector(tl_node_id),
             "some but not all treelite leaves have leaf_vector()");
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_algo");
  };
}

void node2fil_dense(std::vector<dense_node_t>* pnodes, int root, int cur,
                    const tl::Tree& tree, int node_id,
                    const forest_params_t& forest_params) {
  if (tree.IsLeaf(node_id)) {
    node_init(&(*pnodes)[root + cur], val_t{.f = NAN}, NAN, 0, false, true);
    tl2fil_leaf_payload(&(*pnodes)[root + cur], tree, node_id, forest_params);
    return;
  }

  // inner node
  ASSERT(tree.SplitType(node_id) == tl::SplitFeatureType::kNumerical,
         "only numerical split nodes are supported");
  int tl_left = tree.LeftChild(node_id), tl_right = tree.RightChild(node_id);
  bool default_left = tree.DefaultLeft(node_id);
  float threshold = tree.Threshold(node_id);
  adjust_threshold(&threshold, &tl_left, &tl_right, &default_left,
                   tree.ComparisonOp(node_id));
  node_init(&(*pnodes)[root + cur], val_t{.f = 0}, threshold,
            tree.SplitIndex(node_id), default_left, false);
  int left = 2 * cur + 1;
  node2fil_dense(pnodes, root, left, tree, tl_left, forest_params);
  node2fil_dense(pnodes, root, left + 1, tree, tl_right, forest_params);
}

void tree2fil_dense(std::vector<dense_node_t>* pnodes, int root,
                    const tl::Tree& tree,
                    const forest_params_t& forest_params) {
  node2fil_dense(pnodes, root, 0, tree, tree_root(tree), forest_params);
}

template <typename fil_node_t>
int tree2fil_sparse(std::vector<fil_node_t>* pnodes, const tl::Tree& tree,
                    const forest_params_t& forest_params) {
  typedef std::pair<int, int> pair_t;
  std::stack<pair_t> stack;
  int root = pnodes->size();
  pnodes->push_back(fil_node_t());
  stack.push(pair_t(tree_root(tree), 0));
  while (!stack.empty()) {
    const pair_t& top = stack.top();
    int node_id = top.first;
    int cur = top.second;
    stack.pop();

    while (!tree.IsLeaf(node_id)) {
      // inner node
      ASSERT(tree.SplitType(node_id) == tl::SplitFeatureType::kNumerical,
             "only numerical split nodes are supported");
      // tl_left and tl_right are indices of the children in the treelite tree
      // (stored  as an array of nodes)
      int tl_left = tree.LeftChild(node_id),
          tl_right = tree.RightChild(node_id);
      bool default_left = tree.DefaultLeft(node_id);
      float threshold = tree.Threshold(node_id);
      adjust_threshold(&threshold, &tl_left, &tl_right, &default_left,
                       tree.ComparisonOp(node_id));

      // reserve space for child nodes
      // left is the offset of the left child node relative to the tree root
      // in the array of all nodes of the FIL sparse forest
      int left = pnodes->size() - root;
      pnodes->push_back(fil_node_t());
      pnodes->push_back(fil_node_t());
      node_init_inline(&(*pnodes)[root + cur], val_t{.f = 0}, threshold,
                       tree.SplitIndex(node_id), default_left, false, left);

      // push child nodes into the stack
      stack.push(pair_t(tl_right, left + 1));
      //stack.push(pair_t(tl_left, left));
      node_id = tl_left;
      cur = left;
    }

    // leaf node
    node_init_inline(&(*pnodes)[root + cur], val_t{.f = NAN}, NAN, 0, false,
                     true, 0);
    tl2fil_leaf_payload(&(*pnodes)[root + cur], tree, node_id, forest_params);
  }

  return root;
}

size_t tl_leaf_vector_size(const tl::Model& model) {
  const tl::Tree& tree = model.trees[0];
  int node_key;
  for (node_key = tree_root(tree); !tree.IsLeaf(node_key);
       node_key = tree.RightChild(node_key))
    ;
  if (tree.HasLeafVector(node_key)) return tree.LeafVector(node_key).size();
  return 0;
}

// tl2fil_common is the part of conversion from a treelite model
// common for dense and sparse forests
void tl2fil_common(forest_params_t* params, const tl::Model& model,
                   const treelite_params_t* tl_params) {
  // fill in forest-indendent params
  params->algo = tl_params->algo;
  params->threshold = tl_params->threshold;

  // fill in forest-dependent params
  params->depth = max_depth(model);  // also checks for cycles

  const tl::ModelParam& param = model.param;

  // assuming either all leaves use the .leaf_vector() or all leaves use .leaf_value()
  size_t leaf_vec_size = tl_leaf_vector_size(model);
  std::string pred_transform(param.pred_transform);
  if (leaf_vec_size > 0) {
    ASSERT(leaf_vec_size == model.num_output_group,
           "treelite model inconsistent");
    params->num_classes = leaf_vec_size;
    params->leaf_algo = leaf_algo_t::CATEGORICAL_LEAF;

    ASSERT(tl_params->output_class,
           "output_class==true is required for multi-class models");

    ASSERT(
      pred_transform == "max_index" || pred_transform == "identity_multiclass",
      "only max_index and identity_multiclass values of pred_transform "
      "are supported for multi-class models");

  } else {
    if (model.num_output_group > 1) {
      params->num_classes = model.num_output_group;
      ASSERT(tl_params->output_class,
             "output_class==true is required for multi-class models");
      ASSERT(pred_transform == "sigmoid" || pred_transform == "identity" ||
               pred_transform == "max_index" || pred_transform == "softmax" ||
               pred_transform == "multiclass_ova",
             "only sigmoid, identity, max_index, multiclass_ova and softmax "
             "values of pred_transform are supported for xgboost-style "
             "multi-class classification models.");
      // this function should not know how many threads per block will be used
      params->leaf_algo = leaf_algo_t::GROVE_PER_CLASS;
    } else {
      params->num_classes = tl_params->output_class ? 2 : 1;
      ASSERT(pred_transform == "sigmoid" || pred_transform == "identity",
             "only sigmoid and identity values of pred_transform "
             "are supported for binary classification and regression models.");
      params->leaf_algo = leaf_algo_t::FLOAT_UNARY_BINARY;
    }
  }

  params->num_cols = model.num_feature;

  ASSERT(param.sigmoid_alpha == 1.0f, "sigmoid_alpha not supported");
  params->global_bias = param.global_bias;
  params->output = output_t::RAW;
  /** output_t::CLASS denotes using a threshold in FIL, when
      predict_proba == false. For all multiclass models, the best class is
      selected using argmax instead. This happens when either
      leaf_algo == CATEGORICAL_LEAF or num_classes > 2.
  **/
  if (tl_params->output_class && params->leaf_algo != CATEGORICAL_LEAF &&
      params->num_classes <= 2) {
    params->output = output_t(params->output | output_t::CLASS);
  }
  // "random forest" in treelite means tree output averaging
  if (model.random_forest_flag) {
    params->output = output_t(params->output | output_t::AVG);
  }
  if (std::string(param.pred_transform) == "sigmoid") {
    params->output = output_t(params->output | output_t::SIGMOID);
  }
  params->num_trees = model.trees.size();
  params->blocks_per_sm = tl_params->blocks_per_sm;
}

// uses treelite model with additional tl_params to initialize FIL params
// and dense nodes (stored in *pnodes)
void tl2fil_dense(std::vector<dense_node_t>* pnodes, forest_params_t* params,
                  const tl::Model& model, const treelite_params_t* tl_params) {
  tl2fil_common(params, model, tl_params);

  // convert the nodes
  int num_nodes = forest_num_nodes(params->num_trees, params->depth);
  pnodes->resize(num_nodes, dense_node_t{0, 0});
  for (int i = 0; i < model.trees.size(); ++i) {
    tree2fil_dense(pnodes, i * tree_num_nodes(params->depth), model.trees[i],
                   *params);
  }
}

template <typename fil_node_t>
struct tl2fil_sparse_check_t {
  static void check(const tl::Model& model) {
    ASSERT(false,
           "internal error: "
           "only a specialization of this tempalte should be used");
  }
};

template <>
struct tl2fil_sparse_check_t<sparse_node16_t> {
  // no extra check for 16-byte sparse nodes
  static void check(const tl::Model& model) {}
};

template <>
struct tl2fil_sparse_check_t<sparse_node8_t> {
  static const int MAX_FEATURES = 1 << sparse_node8::FID_NUM_BITS;
  static const int MAX_TREE_NODES = (1 << sparse_node8::LEFT_NUM_BITS) - 1;
  static void check(const tl::Model& model) {
    // check the number of features
    int num_features = model.num_feature;
    ASSERT(num_features <= MAX_FEATURES,
           "model has %d features, "
           "but only %d supported for 8-byte sparse nodes",
           num_features, MAX_FEATURES);

    // check the number of tree nodes
    const std::vector<tl::Tree>& trees = model.trees;
    for (int i = 0; i < trees.size(); ++i) {
      int num_nodes = trees[i].num_nodes;
      ASSERT(num_nodes <= MAX_TREE_NODES,
             "tree %d has %d nodes, "
             "but only %d supported for 8-byte sparse nodes",
             i, num_nodes, MAX_TREE_NODES);
    }
  }
};

// uses treelite model with additional tl_params to initialize FIL params,
// trees (stored in *ptrees) and sparse nodes (stored in *pnodes)
template <typename fil_node_t>
void tl2fil_sparse(std::vector<int>* ptrees, std::vector<fil_node_t>* pnodes,
                   forest_params_t* params, const tl::Model& model,
                   const treelite_params_t* tl_params) {
  tl2fil_common(params, model, tl_params);
  tl2fil_sparse_check_t<fil_node_t>::check(model);

  // convert the nodes
  for (int i = 0; i < model.trees.size(); ++i) {
    int root = tree2fil_sparse(pnodes, model.trees[i], *params);
    ptrees->push_back(root);
  }
  params->num_nodes = pnodes->size();
}

void init_dense(const raft::handle_t& h, forest_t* pf,
                const dense_node_t* nodes, const forest_params_t* params) {
  check_params(params, true);
  dense_forest* f = new dense_forest;
  f->init(h, nodes, params);
  *pf = f;
}

template <typename fil_node_t>
void init_sparse(const raft::handle_t& h, forest_t* pf, const int* trees,
                 const typename external_node<fil_node_t>::t* nodes,
                 const forest_params_t* params) {
  check_params(params, false);
  sparse_forest<fil_node_t>* f = new sparse_forest<fil_node_t>;
  f->init(h, trees, nodes, params);
  *pf = f;
}

void init_sparse(const raft::handle_t& h, forest_t* pf, const int* trees,
                 const sparse_node16_t* nodes, const forest_params_t* params) {
  init_sparse<sparse_node16>(h, pf, trees, nodes, params);
}

void init_sparse(const raft::handle_t& h, forest_t* pf, const int* trees,
                 const sparse_node8_t* nodes, const forest_params_t* params) {
  init_sparse<sparse_node8>(h, pf, trees, nodes, params);
}

void from_treelite(const raft::handle_t& handle, forest_t* pforest,
                   ModelHandle model, const treelite_params_t* tl_params) {
  storage_type_t storage_type = tl_params->storage_type;
  // build dense trees by default
  const tl::Model& model_ref = *(tl::Model*)model;
  if (storage_type == storage_type_t::AUTO) {
    if (tl_params->algo == algo_t::ALGO_AUTO ||
        tl_params->algo == algo_t::NAIVE) {
      int depth = max_depth(model_ref);
      // max 2**25 dense nodes, 256 MiB dense model size
      const int LOG2_MAX_DENSE_NODES = 25;
      int log2_num_dense_nodes =
        depth + 1 + int(ceil(std::log2(model_ref.trees.size())));
      storage_type = log2_num_dense_nodes > LOG2_MAX_DENSE_NODES
                       ? storage_type_t::SPARSE
                       : storage_type_t::DENSE;
    } else {
      // only dense storage is supported for other algorithms
      storage_type = storage_type_t::DENSE;
    }
  }

  forest_params_t params;
  switch (storage_type) {
    case storage_type_t::DENSE: {
      std::vector<dense_node_t> nodes;
      tl2fil_dense(&nodes, &params, model_ref, tl_params);
      init_dense(handle, pforest, nodes.data(), &params);
      // sync is necessary as nodes is used in init_dense(),
      // but destructed at the end of this function
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      break;
    }
    case storage_type_t::SPARSE: {
      std::vector<int> trees;
      std::vector<sparse_node16_t> nodes;
      tl2fil_sparse(&trees, &nodes, &params, model_ref, tl_params);
      init_sparse<sparse_node16>(handle, pforest, trees.data(), nodes.data(),
                                 &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      break;
    }
    case storage_type_t::SPARSE8: {
      std::vector<int> trees;
      std::vector<sparse_node8_t> nodes;
      tl2fil_sparse(&trees, &nodes, &params, model_ref, tl_params);
      init_sparse<sparse_node8>(handle, pforest, trees.data(), nodes.data(),
                                &params);
      CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
      break;
    }
    default:
      ASSERT(false, "tl_params->sparse must be one of AUTO, DENSE or SPARSE");
  }
}

void free(const raft::handle_t& h, forest_t f) {
  f->free(h);
  delete f;
}

void predict(const raft::handle_t& h, forest_t f, float* preds,
             const float* data, size_t num_rows, bool predict_proba) {
  f->predict(h, preds, data, num_rows, predict_proba);
}

}  // namespace fil
}  // namespace ML
