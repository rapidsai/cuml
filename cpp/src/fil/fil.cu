#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file fil.cu fil.cu implements the forest data types (dense and sparse), including their
creation and prediction (the main inference kernel is defined in infer.cu). */

#include "common.cuh"    // for predict_params, storage, storage
#include "internal.cuh"  // for cat_sets_device_owner, categorical_sets, output_t,

#include <cuml/fil/fil.h>  // for algo_t,

#include <raft/cudart_utils.h>     // for RAFT_CUDA_TRY, hipStream_t,
#include <raft/error.hpp>          // for ASSERT
#include <raft/handle.hpp>         // for handle_t
#include <rmm/device_uvector.hpp>  // for device_uvector
#include <thrust/host_vector.h>    // for host_vector

#include <cmath>    // for expf
#include <cstddef>  // for size_t
#include <cstdint>  // for uint8_t

namespace ML {
namespace fil {

template <typename real_t>
__host__ __device__ real_t sigmoid(real_t x)
{
  return real_t(1) / (real_t(1) + exp(-x));
}

/** performs additional transformations on the array of forest predictions
    (preds) of size n; the transformations are defined by output, and include
    averaging (multiplying by inv_num_trees), adding global_bias (always done),
    sigmoid and applying threshold. in case of complement_proba,
    fills in the complement probability */
template <typename real_t>
__global__ void transform_k(real_t* preds,
                            size_t n,
                            output_t output,
                            real_t inv_num_trees,
                            real_t threshold,
                            real_t global_bias,
                            bool complement_proba)
{
  size_t i = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (i >= n) return;
  if (complement_proba && i % 2 != 0) return;

  real_t result = preds[i];
  if ((output & output_t::AVG) != 0) result *= inv_num_trees;
  result += global_bias;
  if ((output & output_t::SIGMOID) != 0) result = sigmoid(result);
  // will not be done on CATEGORICAL_LEAF because the whole kernel will not run
  if ((output & output_t::CLASS) != 0) { result = result > threshold ? real_t(1) : real_t(0); }
  // sklearn outputs numpy array in 'C' order, with the number of classes being last dimension
  // that is also the default order, so we should use the same one
  if (complement_proba) {
    preds[i]     = real_t(1) - result;
    preds[i + 1] = result;
  } else
    preds[i] = result;
}

// needed to avoid expanding the dispatch template into unresolved
// compute_smem_footprint::run() calls. In infer.cu, we don't export those symbols,
// but rather one symbol for the whole template specialization, as below.
extern template int dispatch_on_fil_template_params(compute_smem_footprint, predict_params);

// forest is the base type for all forests and contains data and methods common
// to both dense and sparse forests
template <typename real_t>
struct forest {
  forest(const raft::handle_t& h) : cat_sets_(h.get_stream()), vector_leaf_(0, h.get_stream()) {}

  void init_shmem_size(int device)
  {
    /// the most shared memory a kernel can request on the GPU in question
    RAFT_CUDA_TRY(
      hipDeviceGetAttribute(&max_shm_, hipDeviceAttributeSharedMemPerBlockOptin, device));
    /* Our GPUs have been growing the shared memory size generation after
       generation. Eventually, a CUDA GPU might come by that supports more
       shared memory that would fit into unsigned 16-bit int. For such a GPU,
       we would have otherwise silently overflowed the index calculation due
       to short division. It would have failed cpp tests, but we might forget
       about this source of bugs, if not for the failing assert. */
    ASSERT(max_shm_ < int(proba_ssp_.sizeof_real) * std::numeric_limits<uint16_t>::max(),
           "internal error: please use a larger type inside"
           " infer_k for column count");
  }

  void init_n_items(int device)
  {
    // searching for the most items per block while respecting the shared
    // memory limits creates a full linear programming problem.
    // solving it in a single equation looks less tractable than this
    for (bool predict_proba : {false, true}) {
      shmem_size_params& ssp_ = predict_proba ? proba_ssp_ : class_ssp_;
      ssp_.predict_proba      = predict_proba;
      shmem_size_params ssp   = ssp_;
      // if n_items was not provided, try from 1 to MAX_N_ITEMS. Otherwise, use as-is.
      int min_n_items = ssp.n_items == 0 ? 1 : ssp.n_items;
      int max_n_items =
        ssp.n_items == 0 ? (algo_ == algo_t::BATCH_TREE_REORG ? MAX_N_ITEMS : 1) : ssp.n_items;
      for (bool cols_in_shmem : {false, true}) {
        ssp.cols_in_shmem = cols_in_shmem;
        for (ssp.n_items = min_n_items; ssp.n_items <= max_n_items; ++ssp.n_items) {
          ssp.shm_sz = dispatch_on_fil_template_params(compute_smem_footprint(), ssp);
          if (ssp.shm_sz < max_shm_) ssp_ = ssp;
        }
      }
      ASSERT(max_shm_ >= ssp_.shm_sz,
             "FIL out of shared memory. Perhaps the maximum number of \n"
             "supported classes is exceeded? 5'000 would still be safe.");
    }
  }

  void init_fixed_block_count(int device, int blocks_per_sm)
  {
    int max_threads_per_sm, sm_count;
    RAFT_CUDA_TRY(
      hipDeviceGetAttribute(&max_threads_per_sm, hipDeviceAttributeMaxThreadsPerMultiProcessor, device));
    blocks_per_sm = std::min(blocks_per_sm, max_threads_per_sm / FIL_TPB);
    RAFT_CUDA_TRY(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device));
    fixed_block_count_ = blocks_per_sm * sm_count;
  }

  void init_common(const raft::handle_t& h,
                   const categorical_sets& cat_sets,
                   const std::vector<real_t>& vector_leaf,
                   const forest_params_t* params)
  {
    depth_                           = params->depth;
    num_trees_                       = params->num_trees;
    algo_                            = params->algo;
    output_                          = params->output;
    threshold_                       = static_cast<real_t>(params->threshold);
    global_bias_                     = static_cast<real_t>(params->global_bias);
    proba_ssp_.n_items               = params->n_items;
    proba_ssp_.log2_threads_per_tree = log2(params->threads_per_tree);
    proba_ssp_.leaf_algo             = params->leaf_algo;
    proba_ssp_.num_cols              = params->num_cols;
    proba_ssp_.num_classes           = params->num_classes;
    proba_ssp_.cats_present          = cat_sets.cats_present();
    proba_ssp_.sizeof_real           = sizeof(real_t);
    class_ssp_                       = proba_ssp_;

    int device          = h.get_device();
    hipStream_t stream = h.get_stream();
    init_shmem_size(device);
    init_n_items(device);  // n_items takes priority over blocks_per_sm
    init_fixed_block_count(device, params->blocks_per_sm);

    // vector leaf
    if (!vector_leaf.empty()) {
      vector_leaf_.resize(vector_leaf.size() * sizeof(real_t), stream);

      RAFT_CUDA_TRY(hipMemcpyAsync(vector_leaf_.data(),
                                    vector_leaf.data(),
                                    vector_leaf.size() * sizeof(real_t),
                                    hipMemcpyHostToDevice,
                                    stream));
    }

    // categorical features
    cat_sets_ = cat_sets_device_owner(cat_sets, stream);
  }

  virtual void infer(predict_params params, hipStream_t stream) = 0;

  void predict(
    const raft::handle_t& h, real_t* preds, const real_t* data, size_t num_rows, bool predict_proba)
  {
    // Initialize prediction parameters.
    predict_params params(predict_proba ? proba_ssp_ : class_ssp_);
    params.algo     = algo_;
    params.preds    = preds;
    params.data     = data;
    params.num_rows = num_rows;
    // ignored unless predict_proba is true and algo is GROVE_PER_CLASS
    params.transform = output_;
    // fixed_block_count_ == 0 means the number of thread blocks is
    // proportional to the number of rows
    params.num_blocks = fixed_block_count_;

    /**
    The binary classification / regression (FLOAT_UNARY_BINARY) predict_proba() works as follows
      (always 2 outputs):
    RAW: output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error
    write the output of the previous stages and its complement

    The binary classification / regression (FLOAT_UNARY_BINARY) predict() works as follows
      (always 1 output):
    RAW (no values set): output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: apply threshold (equivalent to choosing best class)
    SOFTMAX is set: error

    The multi-class classification / regression (CATEGORICAL_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    SOFTMAX is set: error

    The multi-class classification / regression (CATEGORICAL_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest probability, else output label
    0. SOFTMAX is set: error All other flags (AVG, SIGMOID, CLASS) are ignored

    The multi-class classification / regression (GROVE_PER_CLASS) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid; if SOFTMAX is also set: error
    CLASS is set: ignored
    SOFTMAX is set: softmax is applied after averaging and global_bias

    The multi-class classification / regression (GROVE_PER_CLASS) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS, SOFTMAX) are ignored

    The multi-class classification / regression (VECTOR_LEAF) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid; if SOFTMAX is also set: error
    CLASS is set: ignored
    SOFTMAX is set: softmax is applied after averaging and global_bias
    All other flags (SIGMOID, CLASS, SOFTMAX) are ignored

    The multi-class classification / regression (VECTOR_LEAF) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest margin,
      equal margins resolved in favor of smaller label integer
    All other flags (AVG, SIGMOID, CLASS, SOFTMAX) are ignored
    */
    output_t ot = output_;
    // Treelite applies bias before softmax, but we do after.
    // Simulating treelite order, which cancels out bias.
    // If non-proba prediction used, it still will not matter
    // for the same reason softmax will not.
    real_t global_bias    = (ot & output_t::SOFTMAX) != 0 ? real_t(0) : global_bias_;
    bool complement_proba = false, do_transform;

    if (predict_proba) {
      // no threshold on probabilities
      ot = output_t(ot & ~output_t::CLASS);

      switch (params.leaf_algo) {
        case leaf_algo_t::FLOAT_UNARY_BINARY:
          params.num_outputs = 2;
          complement_proba   = true;
          do_transform       = true;
          break;
        case leaf_algo_t::GROVE_PER_CLASS:
          // for GROVE_PER_CLASS, averaging happens in infer_k
          ot                 = output_t(ot & ~output_t::AVG);
          params.num_outputs = params.num_classes;
          do_transform =
            (ot != output_t::RAW && ot != output_t::SOFTMAX) || global_bias != real_t(0);
          break;
        case leaf_algo_t::CATEGORICAL_LEAF:
          params.num_outputs = params.num_classes;
          do_transform       = ot != output_t::RAW || global_bias_ != real_t(0);
          break;
        case leaf_algo_t::VECTOR_LEAF:
          // for VECTOR_LEAF, averaging happens in infer_k
          ot                 = output_t(ot & ~output_t::AVG);
          params.num_outputs = params.num_classes;
          do_transform =
            (ot != output_t::RAW && ot != output_t::SOFTMAX) || global_bias != real_t(0);
          break;
        default: ASSERT(false, "internal error: predict: invalid leaf_algo %d", params.leaf_algo);
      }
    } else {
      if (params.leaf_algo == leaf_algo_t::FLOAT_UNARY_BINARY) {
        do_transform = ot != output_t::RAW || global_bias_ != real_t(0);
      } else {
        // GROVE_PER_CLASS, CATEGORICAL_LEAF: moot since choosing best class and
        // all transforms are monotonic. also, would break current code
        do_transform = false;
      }
      params.num_outputs = 1;
    }

    // Predict using the forest.
    hipStream_t stream = h.get_stream();
    infer(params, stream);

    if (do_transform) {
      size_t num_values_to_transform = (size_t)num_rows * (size_t)params.num_outputs;
      transform_k<<<raft::ceildiv(num_values_to_transform, (size_t)FIL_TPB), FIL_TPB, 0, stream>>>(
        preds,
        num_values_to_transform,
        ot,
        num_trees_ > 0 ? (real_t(1) / num_trees_) : real_t(1),
        threshold_,
        global_bias,
        complement_proba);
      RAFT_CUDA_TRY(hipPeekAtLastError());
    }
  }

  virtual void free(const raft::handle_t& h)
  {
    cat_sets_.release();
    vector_leaf_.release();
  }

  virtual ~forest() {}

  int num_trees_         = 0;
  int depth_             = 0;
  algo_t algo_           = algo_t::NAIVE;
  output_t output_       = output_t::RAW;
  int fixed_block_count_ = 0;
  int max_shm_           = 0;
  real_t threshold_      = 0.5;
  real_t global_bias_    = 0;
  shmem_size_params class_ssp_;
  shmem_size_params proba_ssp_;
  // vector_leaf_ is only used if {class,proba}_ssp_.leaf_algo is VECTOR_LEAF,
  // otherwise it is empty
  rmm::device_uvector<real_t> vector_leaf_;
  cat_sets_device_owner cat_sets_;
};

template <typename storage_type>
struct opt_into_arch_dependent_shmem : dispatch_functor<void> {
  const int max_shm;
  opt_into_arch_dependent_shmem(int max_shm_) : max_shm(max_shm_) {}

  template <typename KernelParams = KernelTemplateParams<>>
  void run(predict_params p)
  {
    auto kernel = infer_k<KernelParams::N_ITEMS,
                          KernelParams::LEAF_ALGO,
                          KernelParams::COLS_IN_SHMEM,
                          KernelParams::CATS_SUPPORTED,
                          storage_type>;
    // p.shm_sz might be > max_shm or < MAX_SHM_STD, but we should not check for either, because
    // we don't run on both proba_ssp_ and class_ssp_ (only class_ssp_). This should be quick.
    RAFT_CUDA_TRY(
      hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, max_shm));
  }
};

template <typename real_t>
struct dense_forest<dense_node<real_t>> : forest<real_t> {
  using node_t = dense_node<real_t>;
  dense_forest(const raft::handle_t& h) : forest<real_t>(h), nodes_(0, h.get_stream()) {}

  void transform_trees(const node_t* nodes)
  {
    /* Populate node information:
       For each tree, the nodes are still stored in the breadth-first,
       left-to-right order. However, instead of storing the nodes of the same
       tree adjacently, it uses a different layout. In this layout, the roots
       of all trees (node 0) are stored first, followed by left children of
       the roots of all trees (node 1), followed by the right children of the
       roots of all trees (node 2), and so on.
    */
    int global_node = 0;
    for (int tree = 0; tree < this->num_trees_; ++tree) {
      int tree_node = 0;
      // the counters `level` and `branch` are not used for computing node
      // indices, they are only here to highlight the node ordering within
      // each tree
      for (int level = 0; level <= this->depth_; ++level) {
        for (int branch = 0; branch < 1 << level; ++branch) {
          h_nodes_[tree_node * this->num_trees_ + tree] = nodes[global_node];
          ++tree_node;
          ++global_node;
        }
      }
    }
  }

  /// const int* trees is ignored and only provided for compatibility with
  /// sparse_forest<node_t>::init()
  void init(const raft::handle_t& h,
            const categorical_sets& cat_sets,
            const std::vector<real_t>& vector_leaf,
            const int* trees,
            const node_t* nodes,
            const forest_params_t* params)
  {
    this->init_common(h, cat_sets, vector_leaf, params);
    if (this->algo_ == algo_t::NAIVE) this->algo_ = algo_t::BATCH_TREE_REORG;

    int num_nodes = forest_num_nodes(this->num_trees_, this->depth_);
    nodes_.resize(num_nodes, h.get_stream());
    h_nodes_.resize(num_nodes);
    if (this->algo_ == algo_t::NAIVE) {
      std::copy(nodes, nodes + num_nodes, h_nodes_.begin());
    } else {
      transform_trees(nodes);
    }
    RAFT_CUDA_TRY(hipMemcpyAsync(nodes_.data(),
                                  h_nodes_.data(),
                                  num_nodes * sizeof(node_t),
                                  hipMemcpyHostToDevice,
                                  h.get_stream()));

    // predict_proba is a runtime parameter, and opt-in is unconditional
    dispatch_on_fil_template_params(opt_into_arch_dependent_shmem<storage<node_t>>(this->max_shm_),
                                    static_cast<predict_params>(this->class_ssp_));
    // copy must be finished before freeing the host data
    h.sync_stream();
    h_nodes_.clear();
    h_nodes_.shrink_to_fit();
  }

  virtual void infer(predict_params params, hipStream_t stream) override
  {
    storage<node_t> forest(this->cat_sets_.accessor(),
                           this->vector_leaf_.data(),
                           nodes_.data(),
                           this->num_trees_,
                           this->algo_ == algo_t::NAIVE ? tree_num_nodes(this->depth_) : 1,
                           this->algo_ == algo_t::NAIVE ? 1 : this->num_trees_);
    fil::infer(forest, params, stream);
  }

  virtual void free(const raft::handle_t& h) override
  {
    nodes_.release();
    forest<real_t>::free(h);
  }

  rmm::device_uvector<node_t> nodes_;
  thrust::host_vector<node_t> h_nodes_;
};

template <typename node_t>
struct sparse_forest : forest<typename node_t::real_type> {
  using real_type = typename node_t::real_type;

  sparse_forest(const raft::handle_t& h)
    : forest<typename node_t::real_type>(h), trees_(0, h.get_stream()), nodes_(0, h.get_stream())
  {
  }

  void init(const raft::handle_t& h,
            const categorical_sets& cat_sets,
            const std::vector<real_type>& vector_leaf,
            const int* trees,
            const node_t* nodes,
            const forest_params_t* params)
  {
    this->init_common(h, cat_sets, vector_leaf, params);
    if (this->algo_ == algo_t::ALGO_AUTO) this->algo_ = algo_t::NAIVE;
    this->depth_ = 0;  // a placeholder value
    num_nodes_   = params->num_nodes;

    // trees
    trees_.resize(this->num_trees_, h.get_stream());
    RAFT_CUDA_TRY(hipMemcpyAsync(trees_.data(),
                                  trees,
                                  sizeof(int) * this->num_trees_,
                                  hipMemcpyHostToDevice,
                                  h.get_stream()));

    // nodes
    nodes_.resize(num_nodes_, h.get_stream());
    RAFT_CUDA_TRY(hipMemcpyAsync(
      nodes_.data(), nodes, sizeof(node_t) * num_nodes_, hipMemcpyHostToDevice, h.get_stream()));

    // predict_proba is a runtime parameter, and opt-in is unconditional
    dispatch_on_fil_template_params(opt_into_arch_dependent_shmem<storage<node_t>>(this->max_shm_),
                                    static_cast<predict_params>(this->class_ssp_));
  }

  virtual void infer(predict_params params, hipStream_t stream) override
  {
    storage<node_t> forest(this->cat_sets_.accessor(),
                           this->vector_leaf_.data(),
                           trees_.data(),
                           nodes_.data(),
                           this->num_trees_);
    fil::infer(forest, params, stream);
  }

  void free(const raft::handle_t& h) override
  {
    trees_.release();
    nodes_.release();
    forest<real_type>::free(h);
  }

  int num_nodes_ = 0;
  rmm::device_uvector<int> trees_;
  rmm::device_uvector<node_t> nodes_;
};

void check_params(const forest_params_t* params, bool dense)
{
  if (dense) {
    ASSERT(params->depth >= 0, "depth must be non-negative for dense forests");
  } else {
    ASSERT(params->num_nodes >= 0, "num_nodes must be non-negative for sparse forests");
    ASSERT(params->algo == algo_t::NAIVE || params->algo == algo_t::ALGO_AUTO,
           "only ALGO_AUTO and NAIVE algorithms are supported "
           "for sparse forests");
  }
  ASSERT(params->num_trees >= 0, "num_trees must be non-negative");
  ASSERT(params->num_cols >= 0, "num_cols must be non-negative");
  switch (params->algo) {
    case algo_t::ALGO_AUTO:
    case algo_t::NAIVE:
    case algo_t::TREE_REORG:
    case algo_t::BATCH_TREE_REORG: break;
    default: ASSERT(false, "algo should be ALGO_AUTO, NAIVE, TREE_REORG or BATCH_TREE_REORG");
  }
  switch (params->leaf_algo) {
    case leaf_algo_t::FLOAT_UNARY_BINARY:
      if ((params->output & output_t::CLASS) != 0) {
        ASSERT(params->num_classes == 2,
               "only supporting binary"
               " classification using FLOAT_UNARY_BINARY");
      } else {
        ASSERT(params->num_classes == 1,
               "num_classes must be 1 for "
               "regression");
      }
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax does not make sense for leaf_algo == FLOAT_UNARY_BINARY");
      break;
    case leaf_algo_t::GROVE_PER_CLASS:
      ASSERT(params->threads_per_tree == 1, "multiclass not supported with threads_per_tree > 1");
      ASSERT(params->num_classes > 2,
             "num_classes > 2 is required for leaf_algo == GROVE_PER_CLASS");
      ASSERT(params->num_trees % params->num_classes == 0,
             "num_classes must divide num_trees evenly for GROVE_PER_CLASS");
      break;
    case leaf_algo_t::CATEGORICAL_LEAF:
      ASSERT(params->threads_per_tree == 1, "multiclass not supported with threads_per_tree > 1");
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == CATEGORICAL_LEAF");
      ASSERT((params->output & output_t::SOFTMAX) == 0,
             "softmax not supported for leaf_algo == CATEGORICAL_LEAF");
      break;
    case leaf_algo_t::VECTOR_LEAF:
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_algo == VECTOR_LEAF");
      break;
    default:
      ASSERT(false,
             "leaf_algo must be FLOAT_UNARY_BINARY, CATEGORICAL_LEAF"
             " or GROVE_PER_CLASS");
  }
  // output_t::RAW == 0, and doesn't have a separate flag
  if ((params->output & ~output_t::ALL_SET) != 0) {
    ASSERT(false, "output should be a combination of RAW, AVG, SIGMOID, CLASS and SOFTMAX");
  }
  ASSERT(~params->output & (output_t::SIGMOID | output_t::SOFTMAX),
         "combining softmax and sigmoid is not supported");
  ASSERT(params->blocks_per_sm >= 0, "blocks_per_sm must be nonnegative");
  ASSERT(params->n_items >= 0, "n_items must be non-negative");
  ASSERT(params->threads_per_tree > 0, "threads_per_tree must be positive");
  ASSERT(thrust::detail::is_power_of_2(params->threads_per_tree),
         "threads_per_tree must be a power of 2");
  ASSERT(params->threads_per_tree <= FIL_TPB,
         "threads_per_tree must not "
         "exceed block size %d",
         FIL_TPB);
}

/** initializes a forest of any type
 * When fil_node_t == dense_node, const int* trees is ignored
 */
template <typename fil_node_t, typename real_t>
void init(const raft::handle_t& h,
          forest_t<real_t>* pf,
          const categorical_sets& cat_sets,
          const std::vector<real_t>& vector_leaf,
          const int* trees,
          const fil_node_t* nodes,
          const forest_params_t* params)
{
  check_params(params, node_traits<fil_node_t>::IS_DENSE);
  using forest_type = typename node_traits<fil_node_t>::forest;
  forest_type* f    = new forest_type(h);
  f->init(h, cat_sets, vector_leaf, trees, nodes, params);
  *pf = f;
}

// explicit instantiations for init()
template void init<dense_node<float>, float>(const raft::handle_t& h,
                                             forest_t<float>* pf,
                                             const categorical_sets& cat_sets,
                                             const std::vector<float>& vector_leaf,
                                             const int* trees,
                                             const dense_node<float>* nodes,
                                             const forest_params_t* params);
template void init<dense_node<double>, double>(const raft::handle_t& h,
                                               forest_t<double>* pf,
                                               const categorical_sets& cat_sets,
                                               const std::vector<double>& vector_leaf,
                                               const int* trees,
                                               const dense_node<double>* nodes,
                                               const forest_params_t* params);
template void init<sparse_node16<float>, float>(const raft::handle_t& h,
                                                forest_t<float>* pf,
                                                const categorical_sets& cat_sets,
                                                const std::vector<float>& vector_leaf,
                                                const int* trees,
                                                const sparse_node16<float>* nodes,
                                                const forest_params_t* params);
template void init<sparse_node16<double>, double>(const raft::handle_t& h,
                                                  forest_t<double>* pf,
                                                  const categorical_sets& cat_sets,
                                                  const std::vector<double>& vector_leaf,
                                                  const int* trees,
                                                  const sparse_node16<double>* nodes,
                                                  const forest_params_t* params);
template void init<sparse_node8, float>(const raft::handle_t& h,
                                        forest_t<float>* pf,
                                        const categorical_sets& cat_sets,
                                        const std::vector<float>& vector_leaf,
                                        const int* trees,
                                        const sparse_node8* nodes,
                                        const forest_params_t* params);

template <typename real_t>
void free(const raft::handle_t& h, forest_t<real_t> f)
{
  f->free(h);
  delete f;
}

template void free<float>(const raft::handle_t& h, forest_t<float> f);
template void free<double>(const raft::handle_t& h, forest_t<double> f);

template <typename real_t>
void predict(const raft::handle_t& h,
             forest_t<real_t> f,
             real_t* preds,
             const real_t* data,
             size_t num_rows,
             bool predict_proba)
{
  f->predict(h, preds, data, num_rows, predict_proba);
}

template void predict<float>(const raft::handle_t& h,
                             forest_t<float> f,
                             float* preds,
                             const float* data,
                             size_t num_rows,
                             bool predict_proba);
template void predict<double>(const raft::handle_t& h,
                              forest_t<double> f,
                              double* preds,
                              const double* data,
                              size_t num_rows,
                              bool predict_proba);

}  // namespace fil
}  // namespace ML
