#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file fil.cu implements forest inference */

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <treelite/c_api.h>
#include <treelite/tree.h>
#include <algorithm>
#include <cmath>
#include <limits>
#include <stack>
#include <utility>

#include <cuml/fil/fil.h>
#include <cuml/common/cuml_allocator.hpp>
#include "common.cuh"

namespace ML {
namespace fil {

using namespace MLCommon;
namespace tl = treelite;

void dense_node_init(dense_node_t* n, val_t output, float thresh, int fid,
                     bool def_left, bool is_leaf) {
  *n = dense_node(output, thresh, fid, def_left, is_leaf);
}

void dense_node_decode(const dense_node_t* n, val_t* output, float* thresh,
                       int* fid, bool* def_left, bool* is_leaf) {
  dense_node dn(*n);
  *output = dn.output<val_t>();
  *thresh = dn.thresh();
  *fid = dn.fid();
  *def_left = dn.def_left();
  *is_leaf = dn.is_leaf();
}

inline void sparse_node_init_inline(sparse_node_t* node, val_t output,
                                    float thresh, int fid, bool def_left,
                                    bool is_leaf, int left_index) {
  sparse_node n(output, thresh, fid, def_left, is_leaf, left_index);

  *node = sparse_node_t(n, n);
}

void sparse_node_init(sparse_node_t* node, val_t output, float thresh, int fid,
                      bool def_left, bool is_leaf, int left_index) {
  sparse_node_init_inline(node, output, thresh, fid, def_left, is_leaf,
                          left_index);
}

/** sparse_node_decode extracts individual members from node */
void sparse_node_decode(const sparse_node_t* node, val_t* output, float* thresh,
                        int* fid, bool* def_left, bool* is_leaf,
                        int* left_index) {
  dense_node_decode(node, output, thresh, fid, def_left, is_leaf);
  *left_index = sparse_node(*node).left_index();
}

__host__ __device__ float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

/** performs additional transformations on the array of forest predictions
    (preds) of size n; the transformations are defined by output, and include
    averaging (multiplying by inv_num_trees), adding global_bias (always done),
    sigmoid and applying threshold. in case of complement_proba,
    fills in the complement probability */
__global__ void transform_k(float* preds, size_t n, output_t output,
                            float inv_num_trees, float threshold,
                            float global_bias, bool complement_proba) {
  size_t i = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (i >= n) return;
  if (complement_proba && i % 2 != 0) return;

  float result = preds[i];
  if ((output & output_t::AVG) != 0) result *= inv_num_trees;
  result += global_bias;
  if ((output & output_t::SIGMOID) != 0) result = sigmoid(result);
  // will not be done on INT_CLASS_LABEL because the whole kernel will not run
  if ((output & output_t::CLASS) != 0) {
    result = result > threshold ? 1.0f : 0.0f;
  }
  // sklearn outputs numpy array in 'C' order, with the number of classes being last dimension
  // that is also the default order, so we should use the same one
  if (complement_proba) {
    preds[i] = 1.0f - result;
    preds[i + 1] = result;
  } else
    preds[i] = result;
}

struct forest {
  void init_max_shm() {
    int max_shm_std = 48 * 1024;  // 48 KiB
    int device = 0;
    // TODO(canonizer): use cumlHandle for this
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipDeviceGetAttribute(
      &max_shm_, hipDeviceAttributeSharedMemPerBlockOptin, device));
    // TODO(canonizer): use >48KiB shared memory if available
    max_shm_ = std::min(max_shm_, max_shm_std);
  }

  void init_common(const forest_params_t* params) {
    depth_ = params->depth;
    num_trees_ = params->num_trees;
    num_cols_ = params->num_cols;
    algo_ = params->algo;
    output_ = params->output;
    threshold_ = params->threshold;
    global_bias_ = params->global_bias;
    leaf_payload_type_ = params->leaf_payload_type;
    num_classes_ = params->num_classes;
    init_max_shm();
  }

  virtual void infer(predict_params params, hipStream_t stream) = 0;

  void predict(const cumlHandle& h, float* preds, const float* data,
               size_t num_rows, bool predict_proba) {
    // Initialize prediction parameters.
    predict_params params;
    params.num_cols = num_cols_;
    params.algo = algo_;
    params.preds = preds;
    params.data = data;
    params.num_rows = num_rows;
    params.max_shm = max_shm_;
    params.num_classes = num_classes_;
    params.leaf_payload_type = leaf_payload_type_;

    /**
    The binary classification / regression (FLOAT_SCALAR) predict_proba() works as follows
      (always 2 outputs):
    RAW: output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    write the output of the previous stages and its complement

    The binary classification / regression (FLOAT_SCALAR) predict() works as follows
      (always 1 output):
    RAW (no values set): output the sum of tree predictions
    AVG is set: divide by the number of trees (averaging)
    SIGMOID is set: apply sigmoid
    CLASS is set: apply threshold (equivalent to choosing best class)
    
    The multi-class classification / regression (INT_CLASS_LABEL) predict_proba() works as follows
      (always num_classes outputs):
    RAW (no values set): output class votes
    AVG is set: divide by the number of trees (averaging, output class probability)
    SIGMOID is set: apply sigmoid
    CLASS is set: ignored
    
    The multi-class classification / regression (INT_CLASS_LABEL) predict() works as follows
      (always 1 output):
    RAW (no values set): output the label of the class with highest probability, else output label 0.
    AVG is set: ignored
    SIGMOID is set: ignored
    CLASS is set: ignored
    */
    output_t ot = output_;
    bool complement_proba = false, do_transform = global_bias_ != 0.0f;

    if (leaf_payload_type_ == leaf_value_t::FLOAT_SCALAR) {
      if (predict_proba) {
        params.num_outputs = 2;
        ot = output_t(ot & ~output_t::CLASS);  // no threshold on probabilities
        complement_proba = true;
        do_transform = true;
      } else {
        params.num_outputs = 1;
        if (ot != output_t::RAW) do_transform = true;
      }
    } else if (leaf_payload_type_ == leaf_value_t::INT_CLASS_LABEL) {
      if (predict_proba) {
        params.num_outputs = num_classes_;
        ot = output_t(ot & ~output_t::CLASS);  // no threshold on probabilities
        if (ot != output_t::RAW) do_transform = true;
      } else {
        params.num_outputs = 1;
        // moot since choosing best class and all transforms are monotonic
        // also, would break current code
        do_transform = false;
      }
    }

    // Predict using the forest.
    hipStream_t stream = h.getStream();
    infer(params, stream);

    if (do_transform) {
      size_t num_values_to_transform =
        (size_t)num_rows * (size_t)params.num_outputs;
      transform_k<<<ceildiv(num_values_to_transform, (size_t)FIL_TPB), FIL_TPB,
                    0, stream>>>(preds, num_values_to_transform, ot,
                                 num_trees_ > 0 ? (1.0f / num_trees_) : 1.0f,
                                 threshold_, global_bias_, complement_proba);
      CUDA_CHECK(hipPeekAtLastError());
    }
  }

  virtual void free(const cumlHandle& h) = 0;
  virtual ~forest() {}

  int num_trees_ = 0;
  int depth_ = 0;
  int num_cols_ = 0;
  algo_t algo_ = algo_t::NAIVE;
  int max_shm_ = 0;
  output_t output_ = output_t::RAW;
  float threshold_ = 0.5;
  float global_bias_ = 0;
  leaf_value_t leaf_payload_type_ = leaf_value_t::FLOAT_SCALAR;
  int num_classes_ = 0;
};

struct dense_forest : forest {
  void transform_trees(const dense_node_t* nodes) {
    // populate node information
    for (int i = 0, gid = 0; i < num_trees_; ++i) {
      for (int j = 0, nid = 0; j <= depth_; ++j) {
        for (int k = 0; k < 1 << j; ++k, ++nid, ++gid) {
          h_nodes_[nid * num_trees_ + i] = dense_node(nodes[gid]);
        }
      }
    }
  }

  void init(const cumlHandle& h, const dense_node_t* nodes,
            const forest_params_t* params) {
    init_common(params);
    if (algo_ == algo_t::NAIVE) algo_ = algo_t::BATCH_TREE_REORG;

    int num_nodes = forest_num_nodes(num_trees_, depth_);
    nodes_ = (dense_node*)h.getDeviceAllocator()->allocate(
      sizeof(dense_node) * num_nodes, h.getStream());
    h_nodes_.resize(num_nodes);
    if (algo_ == algo_t::NAIVE) {
      std::copy(nodes, nodes + num_nodes, h_nodes_.begin());
    } else {
      transform_trees(nodes);
    }
    CUDA_CHECK(hipMemcpyAsync(nodes_, h_nodes_.data(),
                               num_nodes * sizeof(dense_node),
                               hipMemcpyHostToDevice, h.getStream()));
    // copy must be finished before freeing the host data
    CUDA_CHECK(hipStreamSynchronize(h.getStream()));
    h_nodes_.clear();
    h_nodes_.shrink_to_fit();
  }

  virtual void infer(predict_params params, hipStream_t stream) override {
    dense_storage forest(nodes_, num_trees_,
                         algo_ == algo_t::NAIVE ? tree_num_nodes(depth_) : 1,
                         algo_ == algo_t::NAIVE ? 1 : num_trees_);
    fil::infer(forest, params, stream);
  }

  virtual void free(const cumlHandle& h) override {
    int num_nodes = forest_num_nodes(num_trees_, depth_);
    h.getDeviceAllocator()->deallocate(nodes_, sizeof(dense_node) * num_nodes,
                                       h.getStream());
  }

  dense_node* nodes_ = nullptr;
  thrust::host_vector<dense_node> h_nodes_;
};

struct sparse_forest : forest {
  void init(const cumlHandle& h, const int* trees, const sparse_node_t* nodes,
            const forest_params_t* params) {
    init_common(params);
    if (algo_ == algo_t::ALGO_AUTO) algo_ = algo_t::NAIVE;
    depth_ = 0;  // a placeholder value
    num_nodes_ = params->num_nodes;

    // trees
    trees_ = (int*)h.getDeviceAllocator()->allocate(sizeof(int) * num_trees_,
                                                    h.getStream());
    CUDA_CHECK(hipMemcpyAsync(trees_, trees, sizeof(int) * num_trees_,
                               hipMemcpyHostToDevice, h.getStream()));

    // nodes
    nodes_ = (sparse_node*)h.getDeviceAllocator()->allocate(
      sizeof(sparse_node) * num_nodes_, h.getStream());
    CUDA_CHECK(hipMemcpyAsync(nodes_, nodes, sizeof(sparse_node) * num_nodes_,
                               hipMemcpyHostToDevice, h.getStream()));
  }

  virtual void infer(predict_params params, hipStream_t stream) override {
    sparse_storage forest(trees_, nodes_, num_trees_);
    fil::infer(forest, params, stream);
  }

  void free(const cumlHandle& h) override {
    h.getDeviceAllocator()->deallocate(trees_, sizeof(int) * num_trees_,
                                       h.getStream());
    h.getDeviceAllocator()->deallocate(nodes_, sizeof(sparse_node) * num_nodes_,
                                       h.getStream());
  }

  int num_nodes_ = 0;
  int* trees_ = nullptr;
  sparse_node* nodes_ = nullptr;
};

void check_params(const forest_params_t* params, bool dense) {
  if (dense) {
    ASSERT(params->depth >= 0, "depth must be non-negative for dense forests");
  } else {
    ASSERT(params->num_nodes >= 0,
           "num_nodes must be non-negative for sparse forests");
    ASSERT(params->algo == algo_t::NAIVE || params->algo == algo_t::ALGO_AUTO,
           "only ALGO_AUTO and NAIVE algorithms are supported "
           "for sparse forests");
  }
  ASSERT(params->num_trees >= 0, "num_trees must be non-negative");
  ASSERT(params->num_cols >= 0, "num_cols must be non-negative");
  switch (params->algo) {
    case algo_t::ALGO_AUTO:
    case algo_t::NAIVE:
    case algo_t::TREE_REORG:
    case algo_t::BATCH_TREE_REORG:
      break;
    default:
      ASSERT(false,
             "algo should be ALGO_AUTO, NAIVE, TREE_REORG or BATCH_TREE_REORG");
  }
  switch (params->leaf_payload_type) {
    case leaf_value_t::FLOAT_SCALAR:
      /* params->num_classes is ignored in this case, since the user might call
         predict_proba() on regression. Hence, no point checking the range of
         an ignored variable */
      break;
    case leaf_value_t::INT_CLASS_LABEL:
      ASSERT(params->num_classes >= 2,
             "num_classes >= 2 is required for "
             "leaf_payload_type == INT_CLASS_LABEL");
      break;
    default:
      ASSERT(false,
             "leaf_payload_type should be FLOAT_SCALAR or INT_CLASS_LABEL");
  }
  // output_t::RAW == 0, and doesn't have a separate flag
  output_t all_set =
    output_t(output_t::AVG | output_t::SIGMOID | output_t::CLASS);
  if ((params->output & ~all_set) != 0) {
    ASSERT(false,
           "output should be a combination of RAW, AVG, SIGMOID and CLASS");
  }
}

// tl_node_at is a checked version of tree[i]
inline const tl::Tree::Node& tl_node_at(const tl::Tree& tree, size_t i) {
  ASSERT(i < tree.num_nodes, "node index out of range");
  return tree[i];
}

int tree_root(const tl::Tree& tree) {
  // find the root
  int root = -1;
  for (int i = 0; i < tree.num_nodes; ++i) {
    if (tl_node_at(tree, i).is_root()) {
      ASSERT(root == -1, "multi-root trees not supported");
      root = i;
    }
  }
  ASSERT(root != -1, "a tree must have a root");
  return root;
}

int max_depth_helper(const tl::Tree& tree, const tl::Tree::Node& node,
                     int limit) {
  if (node.is_leaf()) return 0;
  ASSERT(limit > 0,
         "recursion depth limit reached, might be a cycle in the tree");
  return 1 +
         std::max(
           max_depth_helper(tree, tl_node_at(tree, node.cleft()), limit - 1),
           max_depth_helper(tree, tl_node_at(tree, node.cright()), limit - 1));
}

inline int max_depth(const tl::Tree& tree) {
  // trees of this depth aren't used, so it most likely means bad input data,
  // e.g. cycles in the forest
  const int DEPTH_LIMIT = 500;
  int root_index = tree_root(tree);
  typedef std::pair<const tl::Tree::Node*, int> pair_t;
  std::stack<pair_t> stack;
  stack.push(pair_t(&tl_node_at(tree, root_index), 0));
  int max_depth = 0;
  while (!stack.empty()) {
    const pair_t& pair = stack.top();
    const tl::Tree::Node* node = pair.first;
    int depth = pair.second;
    stack.pop();
    while (!node->is_leaf()) {
      stack.push(pair_t(&tl_node_at(tree, node->cleft()), depth + 1));
      node = &tl_node_at(tree, node->cright());
      depth++;
      ASSERT(depth < DEPTH_LIMIT,
             "depth limit reached, might be a cycle in the tree");
    }
    // only need to update depth for leaves
    max_depth = std::max(max_depth, depth);
  }
  return max_depth;
}

int max_depth(const tl::Model& model) {
  int depth = 0;
  for (const auto& tree : model.trees) depth = std::max(depth, max_depth(tree));
  return depth;
}

inline void adjust_threshold(float* pthreshold, int* tl_left, int* tl_right,
                             bool* default_left, const tl::Tree::Node& node) {
  // in treelite (take left node if val [op] threshold),
  // the meaning of the condition is reversed compared to FIL;
  // thus, "<" in treelite corresonds to comparison ">=" used by FIL
  // https://github.com/dmlc/treelite/blob/master/include/treelite/tree.h#L243
  switch (node.comparison_op()) {
    case tl::Operator::kLT:
      break;
    case tl::Operator::kLE:
      // x <= y is equivalent to x < y', where y' is the next representable float
      *pthreshold =
        std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
      break;
    case tl::Operator::kGT:
      // x > y is equivalent to x >= y', where y' is the next representable float
      // left and right still need to be swapped
      *pthreshold =
        std::nextafterf(*pthreshold, std::numeric_limits<float>::infinity());
    case tl::Operator::kGE:
      // swap left and right
      std::swap(*tl_left, *tl_right);
      *default_left = !*default_left;
      break;
    default:
      ASSERT(false, "only <, >, <= and >= comparisons are supported");
  }
}

/** if the vector consists of zeros and a single one, return the position
for the one (assumed class label). Else, asserts false.
If the vector contains a NAN, asserts false */
int find_class_label_from_one_hot(tl::tl_float* vector, int len) {
  bool found_label = false;
  int out;
  for (int i = 0; i < len; ++i) {
    if (vector[i] == 1.0f) {
      ASSERT(!found_label, "label vector contains multiple 1.0f");
      out = i;
      found_label = true;
    } else {
      ASSERT(vector[i] == 0.0f,
             "label vector contains values other than 0.0 and 1.0");
    }
  }
  ASSERT(found_label, "did not find 1.0f in vector");
  return out;
}

template <typename fil_node_t>
void tl2fil_leaf_payload(fil_node_t* fil_node, const tl::Tree::Node& tl_node,
                         const forest_params_t& forest_params) {
  auto vec = tl_node.leaf_vector();
  switch (forest_params.leaf_payload_type) {
    case leaf_value_t::INT_CLASS_LABEL:
      ASSERT(vec.size() == forest_params.num_classes,
             "inconsistent number of classes in treelite leaves");
      fil_node->val.idx = find_class_label_from_one_hot(&vec[0], vec.size());
      break;
    case leaf_value_t::FLOAT_SCALAR:
      fil_node->val.f = tl_node.leaf_value();
      ASSERT(tl_node.leaf_vector().size() == 0,
             "some but not all treelite leaves have leaf_vector()");
      break;
    default:
      ASSERT(false, "internal error: invalid leaf_payload_type");
  };
}

void node2fil_dense(std::vector<dense_node_t>* pnodes, int root, int cur,
                    const tl::Tree& tree, const tl::Tree::Node& node,
                    const forest_params_t& forest_params) {
  if (node.is_leaf()) {
    dense_node_init(&(*pnodes)[root + cur], val_t{.f = NAN}, NAN, 0, false,
                    true);
    tl2fil_leaf_payload(&(*pnodes)[root + cur], node, forest_params);
    return;
  }

  // inner node
  ASSERT(node.split_type() == tl::SplitFeatureType::kNumerical,
         "only numerical split nodes are supported");
  int tl_left = node.cleft(), tl_right = node.cright();
  bool default_left = node.default_left();
  float threshold = node.threshold();
  adjust_threshold(&threshold, &tl_left, &tl_right, &default_left, node);
  dense_node_init(&(*pnodes)[root + cur], val_t{.f = 0}, threshold,
                  node.split_index(), default_left, false);
  int left = 2 * cur + 1;
  node2fil_dense(pnodes, root, left, tree, tl_node_at(tree, tl_left),
                 forest_params);
  node2fil_dense(pnodes, root, left + 1, tree, tl_node_at(tree, tl_right),
                 forest_params);
}

void tree2fil_dense(std::vector<dense_node_t>* pnodes, int root,
                    const tl::Tree& tree,
                    const forest_params_t& forest_params) {
  node2fil_dense(pnodes, root, 0, tree, tl_node_at(tree, tree_root(tree)),
                 forest_params);
}

int tree2fil_sparse(std::vector<sparse_node_t>* pnodes, const tl::Tree& tree,
                    const forest_params_t& forest_params) {
  typedef std::pair<const tl::Tree::Node*, int> pair_t;
  std::stack<pair_t> stack;
  int root = pnodes->size();
  pnodes->push_back(sparse_node_t());
  stack.push(pair_t(&tl_node_at(tree, tree_root(tree)), 0));
  while (!stack.empty()) {
    const pair_t& top = stack.top();
    const tl::Tree::Node* node = top.first;
    int cur = top.second;
    stack.pop();

    while (!node->is_leaf()) {
      // inner node
      ASSERT(node->split_type() == tl::SplitFeatureType::kNumerical,
             "only numerical split nodes are supported");
      // tl_left and tl_right are indices of the children in the treelite tree
      // (stored  as an array of nodes)
      int tl_left = node->cleft(), tl_right = node->cright();
      bool default_left = node->default_left();
      float threshold = node->threshold();
      adjust_threshold(&threshold, &tl_left, &tl_right, &default_left, *node);

      // reserve space for child nodes
      // left is the offset of the left child node relative to the tree root
      // in the array of all nodes of the FIL sparse forest
      int left = pnodes->size() - root;
      pnodes->push_back(sparse_node_t());
      pnodes->push_back(sparse_node_t());
      sparse_node_init_inline(&(*pnodes)[root + cur], val_t{.f = 0}, threshold,
                              node->split_index(), default_left, false, left);

      // push child nodes into the stack
      stack.push(pair_t(&tl_node_at(tree, tl_right), left + 1));
      //stack.push(pair_t(&tl_node_at(tree, tl_left), left));
      node = &tl_node_at(tree, tl_left);
      cur = left;
    }

    // leaf node
    sparse_node_init(&(*pnodes)[root + cur], val_t{.f = NAN}, NAN, 0, false,
                     true, 0);
    tl2fil_leaf_payload(&(*pnodes)[root + cur], *node, forest_params);
  }

  return root;
}

size_t tl_leaf_vector_size(const tl::Model& model) {
  const tl::Tree& tree = model.trees[0];
  int node_key;
  for (node_key = tree_root(tree); !tl_node_at(tree, node_key).is_leaf();
       node_key = tl_node_at(tree, node_key).cright())
    ;
  const tl::Tree::Node& node = tl_node_at(tree, node_key);
  if (node.has_leaf_vector()) return node.leaf_vector().size();
  return 0;
}

// tl2fil_common is the part of conversion from a treelite model
// common for dense and sparse forests
void tl2fil_common(forest_params_t* params, const tl::Model& model,
                   const treelite_params_t* tl_params) {
  // fill in forest-indendent params
  params->algo = tl_params->algo;
  params->threshold = tl_params->threshold;

  // fill in forest-dependent params
  params->depth = max_depth(model);  // also checks for cycles

  // assuming either all leaves use the .leaf_vector() or all leaves use .leaf_value()
  size_t leaf_vec_size = tl_leaf_vector_size(model);
  if (leaf_vec_size > 0) {
    ASSERT(leaf_vec_size == model.num_output_group,
           "treelite model inconsistent");
    params->num_classes = leaf_vec_size;
    params->leaf_payload_type = leaf_value_t::INT_CLASS_LABEL;
  } else {
    params->leaf_payload_type = leaf_value_t::FLOAT_SCALAR;
    params->num_classes = 0;  // ignored
  }

  params->num_cols = model.num_feature;
  const tl::ModelParam& param = model.param;
  ASSERT(param.sigmoid_alpha == 1.0f, "sigmoid_alpha not supported");
  params->global_bias = param.global_bias;
  params->output = output_t::RAW;
  if (tl_params->output_class) {
    params->output = output_t(params->output | output_t::CLASS);
  }
  // "random forest" in treelite means tree output averaging
  if (model.random_forest_flag) {
    params->output = output_t(params->output | output_t::AVG);
  }
  if (param.pred_transform == "sigmoid") {
    params->output = output_t(params->output | output_t::SIGMOID);
  } else if (param.pred_transform != "identity") {
    ASSERT(false, "%s: unsupported treelite prediction transform",
           param.pred_transform.c_str());
  }
  params->num_trees = model.trees.size();
}

// uses treelite model with additional tl_params to initialize FIL params
// and dense nodes (stored in *pnodes)
void tl2fil_dense(std::vector<dense_node_t>* pnodes, forest_params_t* params,
                  const tl::Model& model, const treelite_params_t* tl_params) {
  tl2fil_common(params, model, tl_params);

  // convert the nodes
  int num_nodes = forest_num_nodes(params->num_trees, params->depth);
  pnodes->resize(num_nodes, dense_node_t{0, 0});
  for (int i = 0; i < model.trees.size(); ++i) {
    tree2fil_dense(pnodes, i * tree_num_nodes(params->depth), model.trees[i],
                   *params);
  }
}

// uses treelite model with additional tl_params to initialize FIL params,
// trees (stored in *ptrees) and sparse nodes (stored in *pnodes)
void tl2fil_sparse(std::vector<int>* ptrees, std::vector<sparse_node_t>* pnodes,
                   forest_params_t* params, const tl::Model& model,
                   const treelite_params_t* tl_params) {
  tl2fil_common(params, model, tl_params);

  // convert the nodes
  for (int i = 0; i < model.trees.size(); ++i) {
    int root = tree2fil_sparse(pnodes, model.trees[i], *params);
    ptrees->push_back(root);
  }
  params->num_nodes = pnodes->size();
}

void init_dense(const cumlHandle& h, forest_t* pf, const dense_node_t* nodes,
                const forest_params_t* params) {
  check_params(params, true);
  dense_forest* f = new dense_forest;
  f->init(h, nodes, params);
  *pf = f;
}

void init_sparse(const cumlHandle& h, forest_t* pf, const int* trees,
                 const sparse_node_t* nodes, const forest_params_t* params) {
  check_params(params, false);
  sparse_forest* f = new sparse_forest;
  f->init(h, trees, nodes, params);
  *pf = f;
}

void from_treelite(const cumlHandle& handle, forest_t* pforest,
                   ModelHandle model, const treelite_params_t* tl_params) {
  storage_type_t storage_type = tl_params->storage_type;
  // build dense trees by default
  const tl::Model& model_ref = *(tl::Model*)model;
  if (storage_type == storage_type_t::AUTO) {
    if (tl_params->algo == algo_t::ALGO_AUTO ||
        tl_params->algo == algo_t::NAIVE) {
      int depth = max_depth(model_ref);
      // max 2**25 dense nodes, 256 MiB dense model size
      const int LOG2_MAX_DENSE_NODES = 25;
      int log2_num_dense_nodes =
        depth + 1 + int(ceil(std::log2(model_ref.trees.size())));
      storage_type = log2_num_dense_nodes > LOG2_MAX_DENSE_NODES
                       ? storage_type_t::SPARSE
                       : storage_type_t::DENSE;
    } else {
      // only dense storage is supported for other algorithms
      storage_type = storage_type_t::DENSE;
    }
  }

  forest_params_t params;
  switch (storage_type) {
    case storage_type_t::DENSE: {
      std::vector<dense_node_t> nodes;
      tl2fil_dense(&nodes, &params, model_ref, tl_params);
      init_dense(handle, pforest, nodes.data(), &params);
      // sync is necessary as nodes is used in init_dense(),
      // but destructed at the end of this function
      CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
      break;
    }
    case storage_type_t::SPARSE: {
      std::vector<int> trees;
      std::vector<sparse_node_t> nodes;
      tl2fil_sparse(&trees, &nodes, &params, model_ref, tl_params);
      init_sparse(handle, pforest, trees.data(), nodes.data(), &params);
      CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
      break;
    }
    default:
      ASSERT(false, "tl_params->sparse must be one of AUTO, DENSE or SPARSE");
  }
}

void free(const cumlHandle& h, forest_t f) {
  f->free(h);
  delete f;
}

void predict(const cumlHandle& h, forest_t f, float* preds, const float* data,
             size_t num_rows, bool predict_proba) {
  f->predict(h, preds, data, num_rows, predict_proba);
}

}  // namespace fil
}  // namespace ML
