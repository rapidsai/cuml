#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020-2024, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include "node.cuh"

#include <cuml/common/utils.hpp>

namespace cuml {
namespace genetic {

const int node::kInvalidFeatureId = -1;

node::node() {}

node::node(node::type ft) : t(ft)
{
  ASSERT(is_nonterminal(), "node: ctor with `type` argument expects functions type only!");
  u.fid = kInvalidFeatureId;
}

node::node(int fid) : t(node::type::variable) { u.fid = fid; }

node::node(float val) : t(node::type::constant) { u.val = val; }

node::node(const node& src) : t(src.t), u(src.u) {}

node& node::operator=(const node& src)
{
  t = src.t;
  u = src.u;
  return *this;
}

bool node::is_terminal() const { return detail::is_terminal(t); }

bool node::is_nonterminal() const { return detail::is_nonterminal(t); }

int node::arity() const { return detail::arity(t); }

#define CASE(str, val) \
  if (#val == str) return node::type::val
node::type node::from_str(const std::string& ntype)
{
  CASE(ntype, variable);
  CASE(ntype, constant);
  // note: keep the case statements in alphabetical order under each category of
  // operators.
  // binary operators
  CASE(ntype, add);
  CASE(ntype, atan2);
  CASE(ntype, div);
  CASE(ntype, fdim);
  CASE(ntype, max);
  CASE(ntype, min);
  CASE(ntype, mul);
  CASE(ntype, pow);
  CASE(ntype, sub);
  // unary operators
  CASE(ntype, abs);
  CASE(ntype, acos);
  CASE(ntype, asin);
  CASE(ntype, atan);
  CASE(ntype, acosh);
  CASE(ntype, asinh);
  CASE(ntype, atanh);
  CASE(ntype, cbrt);
  CASE(ntype, cos);
  CASE(ntype, cosh);
  CASE(ntype, cube);
  CASE(ntype, exp);
  CASE(ntype, inv);
  CASE(ntype, log);
  CASE(ntype, neg);
  CASE(ntype, rcbrt);
  CASE(ntype, rsqrt);
  CASE(ntype, sq);
  CASE(ntype, sqrt);
  CASE(ntype, sin);
  CASE(ntype, sinh);
  CASE(ntype, tan);
  CASE(ntype, tanh);
  ASSERT(false, "node::from_str: Bad type passed '%s'!", ntype.c_str());
}
#undef CASE

}  // namespace genetic
}  // namespace cuml
