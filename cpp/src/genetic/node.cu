#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/common/utils.hpp>
#include "node.cuh"

namespace cuml {
namespace genetic {

const int node::kInvalidFeatureId = -1;

node::node(node::type ft) : t(ft) {
  ASSERT(is_nonterminal(),
         "node: ctor with `type` argument expects functions type only!");
  u.fid = kInvalidFeatureId;
}

node::node(int fid) : t(node::type::variable) { u.fid = fid; }

node::node(float val) : t(node::type::constant) { u.val = val; }

node::node(const node& src) : t(src.t), u(src.u) {}

node& node::operator=(const node& src) {
  t = src.t;
  u = src.u;
  return *this;
}

bool node::is_terminal() const { return detail::is_terminal(t); }

bool node::is_nonterminal() const { return detail::is_nonterminal(t); }

int node::arity() const { return detail::arity(t); }

#define CASE(str, val) \
  if (#val == str) return node::type::val
node::type node::from_str(const std::string& ntype) {
  CASE(ntype, variable);
  CASE(ntype, constant);
  // note: keep the case statements in alphabetical order under each category of
  // operators.
  // binary operators
  CASE(ntype, add);
  CASE(ntype, atan2);
  CASE(ntype, div);
  CASE(ntype, fdim);
  CASE(ntype, max);
  CASE(ntype, min);
  CASE(ntype, mul);
  CASE(ntype, pow);
  CASE(ntype, sub);
  // unary operators
  CASE(ntype, abs);
  CASE(ntype, acos);
  CASE(ntype, asin);
  CASE(ntype, atan);
  CASE(ntype, acosh);
  CASE(ntype, asinh);
  CASE(ntype, atanh);
  CASE(ntype, cbrt);
  CASE(ntype, cos);
  CASE(ntype, cosh);
  CASE(ntype, cube);
  CASE(ntype, exp);
  CASE(ntype, inv);
  CASE(ntype, log);
  CASE(ntype, neg);
  CASE(ntype, rcbrt);
  CASE(ntype, rsqrt);
  CASE(ntype, sq);
  CASE(ntype, sqrt);
  CASE(ntype, sin);
  CASE(ntype, sinh);
  CASE(ntype, tan);
  CASE(ntype, tanh);
  ASSERT(false, "node::from_str: Bad type passed '%s'!", ntype.c_str());
}
#undef CASE

}  // namespace genetic
}  // namespace cuml
