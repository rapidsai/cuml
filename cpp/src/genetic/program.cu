#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/common/logger.hpp>
#include <cuml/genetic/node.h>
#include <cuml/genetic/program.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/unary_op.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <numeric>
#include <random>
#include <stack>

#include "constants.h"
#include "fitness.cuh"
#include "node.cuh"
#include "reg_stack.cuh"

namespace cuml {
namespace genetic {

/**
 * Execution kernel for a single program. We assume that the input data
 * is stored in column major format.
 */
template <int MaxSize = MAX_STACK_SIZE>
__global__ void execute_kernel(const program_t d_progs,
                               const float* data,
                               float* y_pred,
                               const uint64_t n_rows)
{
  uint64_t pid    = blockIdx.y;                             // current program
  uint64_t row_id = blockIdx.x * blockDim.x + threadIdx.x;  // current dataset row

  if (row_id >= n_rows) { return; }

  stack<float, MaxSize> eval_stack;  // Maintain stack only for remaining threads
  program_t curr_p = d_progs + pid;  // Current program

  int end         = curr_p->len - 1;
  node* curr_node = curr_p->nodes + end;

  float res   = 0.0f;
  float in[2] = {0.0f, 0.0f};

  while (end >= 0) {
    if (detail::is_nonterminal(curr_node->t)) {
      int ar = detail::arity(curr_node->t);
      in[0]  = eval_stack.pop();  // Min arity of function is 1
      if (ar > 1) in[1] = eval_stack.pop();
    }
    res = detail::evaluate_node(*curr_node, data, n_rows, row_id, in);
    eval_stack.push(res);
    curr_node--;
    end--;
  }

  // Outputs stored in col-major format
  y_pred[pid * n_rows + row_id] = eval_stack.pop();
}

program::program()
  : len(0),
    depth(0),
    raw_fitness_(0.0f),
    metric(metric_t::mse),
    mut_type(mutation_t::none),
    nodes(nullptr)
{
}

program::~program() { delete[] nodes; }

program::program(const program& src)
  : len(src.len),
    depth(src.depth),
    raw_fitness_(src.raw_fitness_),
    metric(src.metric),
    mut_type(src.mut_type)
{
  nodes = new node[len];
  std::copy(src.nodes, src.nodes + src.len, nodes);
}

program& program::operator=(const program& src)
{
  len          = src.len;
  depth        = src.depth;
  raw_fitness_ = src.raw_fitness_;
  metric       = src.metric;
  mut_type     = src.mut_type;

  // Copy nodes
  delete[] nodes;
  nodes = new node[len];
  std::copy(src.nodes, src.nodes + src.len, nodes);

  return *this;
}

void compute_metric(const raft::handle_t& h,
                    int n_rows,
                    int n_progs,
                    const float* y,
                    const float* y_pred,
                    const float* w,
                    float* score,
                    const param& params)
{
  // Call appropriate metric function based on metric defined in params
  if (params.metric == metric_t::pearson) {
    weightedPearson(h, n_rows, n_progs, y, y_pred, w, score);
  } else if (params.metric == metric_t::spearman) {
    weightedSpearman(h, n_rows, n_progs, y, y_pred, w, score);
  } else if (params.metric == metric_t::mae) {
    meanAbsoluteError(h, n_rows, n_progs, y, y_pred, w, score);
  } else if (params.metric == metric_t::mse) {
    meanSquareError(h, n_rows, n_progs, y, y_pred, w, score);
  } else if (params.metric == metric_t::rmse) {
    rootMeanSquareError(h, n_rows, n_progs, y, y_pred, w, score);
  } else if (params.metric == metric_t::logloss) {
    logLoss(h, n_rows, n_progs, y, y_pred, w, score);
  } else {
    // This should not be reachable
  }
}

void execute(const raft::handle_t& h,
             const program_t& d_progs,
             const int n_rows,
             const int n_progs,
             const float* data,
             float* y_pred)
{
  hipStream_t stream = h.get_stream();

  dim3 blks(raft::ceildiv(n_rows, GENE_TPB), n_progs, 1);
  execute_kernel<<<blks, GENE_TPB, 0, stream>>>(d_progs, data, y_pred, (uint64_t)n_rows);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

void find_fitness(const raft::handle_t& h,
                  program_t& d_prog,
                  float* score,
                  const param& params,
                  const int n_rows,
                  const float* data,
                  const float* y,
                  const float* sample_weights)
{
  hipStream_t stream = h.get_stream();

  // Compute predicted values
  rmm::device_uvector<float> y_pred(n_rows, stream);
  execute(h, d_prog, n_rows, 1, data, y_pred.data());

  // Compute error
  compute_metric(h, n_rows, 1, y, y_pred.data(), sample_weights, score, params);
}

void find_batched_fitness(const raft::handle_t& h,
                          int n_progs,
                          program_t& d_progs,
                          float* score,
                          const param& params,
                          const int n_rows,
                          const float* data,
                          const float* y,
                          const float* sample_weights)
{
  hipStream_t stream = h.get_stream();

  rmm::device_uvector<float> y_pred((uint64_t)n_rows * (uint64_t)n_progs, stream);
  execute(h, d_progs, n_rows, n_progs, data, y_pred.data());

  // Compute error
  compute_metric(h, n_rows, n_progs, y, y_pred.data(), sample_weights, score, params);
}

void set_fitness(const raft::handle_t& h,
                 program_t& d_prog,
                 program& h_prog,
                 const param& params,
                 const int n_rows,
                 const float* data,
                 const float* y,
                 const float* sample_weights)
{
  hipStream_t stream = h.get_stream();

  rmm::device_uvector<float> score(1, stream);

  find_fitness(h, d_prog, score.data(), params, n_rows, data, y, sample_weights);

  // Update host and device score for program
  RAFT_CUDA_TRY(hipMemcpyAsync(
    &d_prog[0].raw_fitness_, score.data(), sizeof(float), hipMemcpyDeviceToDevice, stream));
  h_prog.raw_fitness_ = score.front_element(stream);
}

void set_batched_fitness(const raft::handle_t& h,
                         int n_progs,
                         program_t& d_progs,
                         std::vector<program>& h_progs,
                         const param& params,
                         const int n_rows,
                         const float* data,
                         const float* y,
                         const float* sample_weights)
{
  hipStream_t stream = h.get_stream();

  rmm::device_uvector<float> score(n_progs, stream);

  find_batched_fitness(h, n_progs, d_progs, score.data(), params, n_rows, data, y, sample_weights);

  // Update scores on host and device
  // TODO: Find a way to reduce the number of implicit memory transfers
  for (auto i = 0; i < n_progs; ++i) {
    RAFT_CUDA_TRY(hipMemcpyAsync(&d_progs[i].raw_fitness_,
                                  score.element_ptr(i),
                                  sizeof(float),
                                  hipMemcpyDeviceToDevice,
                                  stream));
    h_progs[i].raw_fitness_ = score.element(i, stream);
  }
}

float get_fitness(const program& prog, const param& params)
{
  int crit      = params.criterion();
  float penalty = params.parsimony_coefficient * prog.len * (2 * crit - 1);
  return (prog.raw_fitness_ - penalty);
}

/**
 * @brief Get a random subtree of the current program nodes (on CPU)
 *
 * @param pnodes  AST represented as a list of nodes
 * @param len     The total number of nodes in the AST
 * @param rng     Random number generator for subtree selection
 * @return A tuple [first,last) which contains the required subtree
 */
std::pair<int, int> get_subtree(node* pnodes, int len, std::mt19937& rng)
{
  int start, end;
  start = end = 0;

  // Specify RNG
  std::uniform_real_distribution<float> dist_uniform(0.0f, 1.0f);
  float bound = dist_uniform(rng);

  // Specify subtree start probs acc to Koza's selection approach
  std::vector<float> node_probs(len, 0.1);
  float sum = 0.1 * len;

  for (int i = 0; i < len; ++i) {
    if (pnodes[i].is_nonterminal()) {
      node_probs[i] = 0.9;
      sum += 0.8;
    }
  }

  // Normalize vector
  for (int i = 0; i < len; ++i) {
    node_probs[i] /= sum;
  }

  // Compute cumulative sum
  std::partial_sum(node_probs.begin(), node_probs.end(), node_probs.begin());

  start = std::lower_bound(node_probs.begin(), node_probs.end(), bound) - node_probs.begin();
  end   = start;

  // Iterate until all function arguments are satisfied in current subtree
  int num_args = 1;
  while (num_args > end - start) {
    node curr;
    curr = pnodes[end];
    if (curr.is_nonterminal()) num_args += curr.arity();
    ++end;
  }

  return std::make_pair(start, end);
}

int get_depth(const program& p_out)
{
  int depth = 0;
  std::stack<int> arity_stack;
  for (auto i = 0; i < p_out.len; ++i) {
    node curr(p_out.nodes[i]);

    // Update depth
    int sz = arity_stack.size();
    depth  = std::max(depth, sz);

    // Update stack
    if (curr.is_nonterminal()) {
      arity_stack.push(curr.arity());
    } else {
      // Only triggered for a depth 0 node
      if (arity_stack.empty()) break;

      int e = arity_stack.top();
      arity_stack.pop();
      arity_stack.push(e - 1);

      while (arity_stack.top() == 0) {
        arity_stack.pop();
        if (arity_stack.empty()) break;

        e = arity_stack.top();
        arity_stack.pop();
        arity_stack.push(e - 1);
      }
    }
  }

  return depth;
}

void build_program(program& p_out, const param& params, std::mt19937& rng)
{
  // Define data structures needed for tree
  std::stack<int> arity_stack;
  std::vector<node> nodelist;
  nodelist.reserve(1 << (MAX_STACK_SIZE));

  // Specify Distributions with parameters
  std::uniform_int_distribution<int> dist_function(0, params.function_set.size() - 1);
  std::uniform_int_distribution<int> dist_initDepth(params.init_depth[0], params.init_depth[1]);
  std::uniform_int_distribution<int> dist_terminalChoice(0, params.num_features);
  std::uniform_real_distribution<float> dist_constVal(params.const_range[0], params.const_range[1]);
  std::bernoulli_distribution dist_nodeChoice(params.terminalRatio);
  std::bernoulli_distribution dist_coinToss(0.5);

  // Initialize nodes
  int max_depth   = dist_initDepth(rng);
  node::type func = params.function_set[dist_function(rng)];
  node curr_node(func);
  nodelist.push_back(curr_node);
  arity_stack.push(curr_node.arity());

  init_method_t method = params.init_method;
  if (method == init_method_t::half_and_half) {
    // Choose either grow or full for this tree
    bool choice = dist_coinToss(rng);
    method      = choice ? init_method_t::grow : init_method_t::full;
  }

  // Fill tree
  while (!arity_stack.empty()) {
    int depth        = arity_stack.size();
    p_out.depth      = std::max(depth, p_out.depth);
    bool node_choice = dist_nodeChoice(rng);

    if ((node_choice == false || method == init_method_t::full) && depth < max_depth) {
      // Add a function to node list
      curr_node = node(params.function_set[dist_function(rng)]);
      nodelist.push_back(curr_node);
      arity_stack.push(curr_node.arity());
    } else {
      // Add terminal
      int terminal_choice = dist_terminalChoice(rng);
      if (terminal_choice == params.num_features) {
        // Add constant
        float val = dist_constVal(rng);
        curr_node = node(val);
      } else {
        // Add variable
        int fid   = terminal_choice;
        curr_node = node(fid);
      }

      // Modify nodelist
      nodelist.push_back(curr_node);

      // Modify stack
      int e = arity_stack.top();
      arity_stack.pop();
      arity_stack.push(e - 1);
      while (arity_stack.top() == 0) {
        arity_stack.pop();
        if (arity_stack.empty()) { break; }

        e = arity_stack.top();
        arity_stack.pop();
        arity_stack.push(e - 1);
      }
    }
  }

  // Set new program parameters - need to do a copy as
  // nodelist will be deleted using RAII semantics
  p_out.nodes = new node[nodelist.size()];
  std::copy(nodelist.begin(), nodelist.end(), p_out.nodes);

  p_out.len          = nodelist.size();
  p_out.metric       = params.metric;
  p_out.raw_fitness_ = 0.0f;
}

void point_mutation(const program& prog, program& p_out, const param& params, std::mt19937& rng)
{
  // deep-copy program
  p_out = prog;

  // Specify RNGs
  std::uniform_real_distribution<float> dist_uniform(0.0f, 1.0f);
  std::uniform_int_distribution<int> dist_terminalChoice(0, params.num_features);
  std::uniform_real_distribution<float> dist_constantVal(params.const_range[0],
                                                         params.const_range[1]);

  // Fill with uniform numbers
  std::vector<float> node_probs(p_out.len);
  std::generate(
    node_probs.begin(), node_probs.end(), [&dist_uniform, &rng] { return dist_uniform(rng); });

  // Mutate nodes
  int len = p_out.len;
  for (int i = 0; i < len; ++i) {
    node curr(prog.nodes[i]);

    if (node_probs[i] < params.p_point_replace) {
      if (curr.is_terminal()) {
        int choice = dist_terminalChoice(rng);

        if (choice == params.num_features) {
          // Add a randomly generated constant
          curr = node(dist_constantVal(rng));
        } else {
          // Add a variable with fid=choice
          curr = node(choice);
        }
      } else if (curr.is_nonterminal()) {
        // Replace current function with another function of the same arity
        int ar = curr.arity();
        // CUML_LOG_DEBUG("Arity is %d, curr function is
        // %d",ar,static_cast<std::underlying_type<node::type>::type>(curr.t));
        std::vector<node::type> fset = params.arity_set.at(ar);
        std::uniform_int_distribution<> dist_fset(0, fset.size() - 1);
        int choice = dist_fset(rng);
        curr       = node(fset[choice]);
      }

      // Update p_out with updated value
      p_out.nodes[i] = curr;
    }
  }
}

void crossover(
  const program& prog, const program& donor, program& p_out, const param& params, std::mt19937& rng)
{
  // Get a random subtree of prog to replace
  std::pair<int, int> prog_slice = get_subtree(prog.nodes, prog.len, rng);
  int prog_start                 = prog_slice.first;
  int prog_end                   = prog_slice.second;

  // Set metric of output program
  p_out.metric = prog.metric;

  // MAX_STACK_SIZE can only handle tree of depth MAX_STACK_SIZE - max(func_arity=2) + 1
  // Thus we continuously hoist the donor subtree.
  // Actual indices in donor
  int donor_start  = 0;
  int donor_end    = donor.len;
  int output_depth = 0;
  int iter         = 0;
  do {
    ++iter;
    // Get donor subtree
    std::pair<int, int> donor_slice =
      get_subtree(donor.nodes + donor_start, donor_end - donor_start, rng);

    // Get indices w.r.t current subspace [donor_start,donor_end)
    int donor_substart = donor_slice.first;
    int donor_subend   = donor_slice.second;

    // Update relative indices to global indices
    donor_substart += donor_start;
    donor_subend += donor_start;

    // Update to new subspace
    donor_start = donor_substart;
    donor_end   = donor_subend;

    // Evolve on current subspace
    p_out.len = (prog_start) + (donor_end - donor_start) + (prog.len - prog_end);
    delete[] p_out.nodes;
    p_out.nodes = new node[p_out.len];

    // Copy slices using std::copy
    std::copy(prog.nodes, prog.nodes + prog_start, p_out.nodes);
    std::copy(donor.nodes + donor_start, donor.nodes + donor_end, p_out.nodes + prog_start);
    std::copy(prog.nodes + prog_end,
              prog.nodes + prog.len,
              p_out.nodes + (prog_start) + (donor_end - donor_start));

    output_depth = get_depth(p_out);
  } while (output_depth >= MAX_STACK_SIZE);

  // Set the depth of the final program
  p_out.depth = output_depth;
}

void subtree_mutation(const program& prog, program& p_out, const param& params, std::mt19937& rng)
{
  // Generate a random program and perform crossover
  program new_program;
  build_program(new_program, params, rng);
  crossover(prog, new_program, p_out, params, rng);
}

void hoist_mutation(const program& prog, program& p_out, const param& params, std::mt19937& rng)
{
  // Replace program subtree with a random sub-subtree

  std::pair<int, int> prog_slice = get_subtree(prog.nodes, prog.len, rng);
  int prog_start                 = prog_slice.first;
  int prog_end                   = prog_slice.second;

  std::pair<int, int> sub_slice = get_subtree(prog.nodes + prog_start, prog_end - prog_start, rng);
  int sub_start                 = sub_slice.first;
  int sub_end                   = sub_slice.second;

  // Update subtree indices to global indices
  sub_start += prog_start;
  sub_end += prog_start;

  p_out.len    = (prog_start) + (sub_end - sub_start) + (prog.len - prog_end);
  p_out.nodes  = new node[p_out.len];
  p_out.metric = prog.metric;

  // Copy node slices using std::copy
  std::copy(prog.nodes, prog.nodes + prog_start, p_out.nodes);
  std::copy(prog.nodes + sub_start, prog.nodes + sub_end, p_out.nodes + prog_start);
  std::copy(prog.nodes + prog_end,
            prog.nodes + prog.len,
            p_out.nodes + (prog_start) + (sub_end - sub_start));

  // Update depth
  p_out.depth = get_depth(p_out);
}

}  // namespace genetic
}  // namespace cuml