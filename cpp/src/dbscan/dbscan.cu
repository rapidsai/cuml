/*
 * Copyright (c) 2018-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/cluster/dbscan.hpp>

#include <raft/cudart_utils.h>
#include "dbscan.cuh"

namespace ML {
namespace Dbscan {

void fit(const raft::handle_t &handle, float *input, int n_rows, int n_cols,
         float eps, int min_pts, int *labels, int *core_sample_indices,
         size_t max_bytes_per_batch, int verbosity, bool opg) {
  if (opg)
    dbscanFitImpl<float, int, true>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
  else
    dbscanFitImpl<float, int, false>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
}

void fit(const raft::handle_t &handle, double *input, int n_rows, int n_cols,
         double eps, int min_pts, int *labels, int *core_sample_indices,
         size_t max_bytes_per_batch, int verbosity, bool opg) {
  if (opg)
    dbscanFitImpl<double, int, true>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
  else
    dbscanFitImpl<double, int, false>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
}

void fit(const raft::handle_t &handle, float *input, int64_t n_rows,
         int64_t n_cols, float eps, int min_pts, int64_t *labels,
         int64_t *core_sample_indices, size_t max_bytes_per_batch,
         int verbosity, bool opg) {
  if (opg)
    dbscanFitImpl<float, int64_t, true>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
  else
    dbscanFitImpl<float, int64_t, false>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
}

void fit(const raft::handle_t &handle, double *input, int64_t n_rows,
         int64_t n_cols, double eps, int min_pts, int64_t *labels,
         int64_t *core_sample_indices, size_t max_bytes_per_batch,
         int verbosity, bool opg) {
  if (opg)
    dbscanFitImpl<double, int64_t, true>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
  else
    dbscanFitImpl<double, int64_t, false>(
      handle, input, n_rows, n_cols, eps, min_pts, labels, core_sample_indices,
      max_bytes_per_batch, handle.get_stream(), verbosity);
}

}  // namespace Dbscan
}  // namespace ML
