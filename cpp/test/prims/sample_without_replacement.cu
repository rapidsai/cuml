/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <cuda_utils.cuh>
#include <random/rng.cuh>
#include <set>
#include <vector>
#include "test_utils.h"

namespace MLCommon {
namespace Random {

// Terminology:
// SWoR - Sample Without Replacement

template <typename T>
struct SWoRInputs {
  int len, sampledLen;
  int largeWeightIndex;
  T largeWeight;
  GeneratorType gtype;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const SWoRInputs<T>& dims) {
  return os;
}

template <typename T>
class SWoRTest : public ::testing::TestWithParam<SWoRInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<SWoRInputs<T>>::GetParam();
    CUDA_CHECK(hipStreamCreate(&stream));
    allocator.reset(new raft::mr::device::default_allocator);
    Rng r(params.seed, params.gtype);
    allocate(in, params.len);
    allocate(wts, params.len);
    allocate(out, params.sampledLen);
    allocate(outIdx, params.sampledLen);
    h_outIdx.resize(params.sampledLen);
    r.uniform(in, params.len, T(-1.0), T(1.0), stream);
    r.uniform(wts, params.len, T(1.0), T(2.0), stream);
    if (params.largeWeightIndex >= 0) {
      updateDevice(wts + params.largeWeightIndex, &params.largeWeight, 1,
                   stream);
    }
    r.sampleWithoutReplacement(out, outIdx, in, wts, params.sampledLen,
                               params.len, allocator, stream);
    updateHost(&(h_outIdx[0]), outIdx, params.sampledLen, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(wts));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(outIdx));
  }

 protected:
  SWoRInputs<T> params;
  T *in, *out, *wts;
  int* outIdx;
  std::vector<int> h_outIdx;
  hipStream_t stream;
  std::shared_ptr<deviceAllocator> allocator;
};

typedef SWoRTest<float> SWoRTestF;
const std::vector<SWoRInputs<float>> inputsf = {
  {1024, 512, -1, 0.f, GenPhilox, 1234ULL},
  {1024, 1024, -1, 0.f, GenPhilox, 1234ULL},
  {1024, 512 + 1, -1, 0.f, GenPhilox, 1234ULL},
  {1024, 1024 - 1, -1, 0.f, GenPhilox, 1234ULL},
  {1024, 512 + 2, -1, 0.f, GenPhilox, 1234ULL},
  {1024, 1024 - 2, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 1, 512, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 1, 1024, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 1, 512 + 1, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 1, 1024 + 1, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 1, 512 + 2, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 1, 1024 - 2, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 2, 512, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 2, 1024, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 2, 512 + 1, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 2, 1024 + 1, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 2, 512 + 2, -1, 0.f, GenPhilox, 1234ULL},
  {1024 + 2, 1024 + 2, -1, 0.f, GenPhilox, 1234ULL},
  {1024, 512, 10, 100000.f, GenPhilox, 1234ULL},

  {1024, 512, -1, 0.f, GenTaps, 1234ULL},
  {1024, 1024, -1, 0.f, GenTaps, 1234ULL},
  {1024, 512 + 1, -1, 0.f, GenTaps, 1234ULL},
  {1024, 1024 - 1, -1, 0.f, GenTaps, 1234ULL},
  {1024, 512 + 2, -1, 0.f, GenTaps, 1234ULL},
  {1024, 1024 - 2, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 1, 512, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 1, 1024, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 1, 512 + 1, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 1, 1024 + 1, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 1, 512 + 2, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 1, 1024 - 2, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 2, 512, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 2, 1024, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 2, 512 + 1, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 2, 1024 + 1, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 2, 512 + 2, -1, 0.f, GenTaps, 1234ULL},
  {1024 + 2, 1024 + 2, -1, 0.f, GenTaps, 1234ULL},
  {1024, 512, 10, 100000.f, GenTaps, 1234ULL},

  {1024, 512, -1, 0.f, GenKiss99, 1234ULL},
  {1024, 1024, -1, 0.f, GenKiss99, 1234ULL},
  {1024, 512 + 1, -1, 0.f, GenKiss99, 1234ULL},
  {1024, 1024 - 1, -1, 0.f, GenKiss99, 1234ULL},
  {1024, 512 + 2, -1, 0.f, GenKiss99, 1234ULL},
  {1024, 1024 - 2, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 1, 512, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 1, 1024, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 1, 512 + 1, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 1, 1024 + 1, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 1, 512 + 2, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 1, 1024 - 2, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 2, 512, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 2, 1024, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 2, 512 + 1, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 2, 1024 + 1, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 2, 512 + 2, -1, 0.f, GenKiss99, 1234ULL},
  {1024 + 2, 1024 + 2, -1, 0.f, GenKiss99, 1234ULL},
  {1024, 512, 10, 100000.f, GenKiss99, 1234ULL},
};

TEST_P(SWoRTestF, Result) {
  std::set<int> occurence;
  for (int i = 0; i < params.sampledLen; ++i) {
    auto val = h_outIdx[i];
    // indices must be in the given range
    ASSERT_TRUE(0 <= val && val < params.len)
      << "out-of-range index @i=" << i << " val=" << val
      << " sampledLen=" << params.sampledLen;
    // indices should not repeat
    ASSERT_TRUE(occurence.find(val) == occurence.end())
      << "repeated index @i=" << i << " idx=" << val;
    occurence.insert(val);
  }
  // if there's a skewed distribution, the top index should correspond to the
  // particular item with a large weight
  if (params.largeWeightIndex >= 0) {
    ASSERT_EQ(h_outIdx[0], params.largeWeightIndex);
  }
}
INSTANTIATE_TEST_CASE_P(SWoRTests, SWoRTestF, ::testing::ValuesIn(inputsf));

typedef SWoRTest<double> SWoRTestD;
const std::vector<SWoRInputs<double>> inputsd = {
  {1024, 512, -1, 0.0, GenPhilox, 1234ULL},
  {1024, 1024, -1, 0.0, GenPhilox, 1234ULL},
  {1024, 512 + 1, -1, 0.0, GenPhilox, 1234ULL},
  {1024, 1024 - 1, -1, 0.0, GenPhilox, 1234ULL},
  {1024, 512 + 2, -1, 0.0, GenPhilox, 1234ULL},
  {1024, 1024 - 2, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 1, 512, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 1, 1024, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 1, 512 + 1, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 1, 1024 + 1, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 1, 512 + 2, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 1, 1024 - 2, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 2, 512, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 2, 1024, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 2, 512 + 1, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 2, 1024 + 1, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 2, 512 + 2, -1, 0.0, GenPhilox, 1234ULL},
  {1024 + 2, 1024 + 2, -1, 0.0, GenPhilox, 1234ULL},
  {1024, 512, 10, 100000.0, GenPhilox, 1234ULL},

  {1024, 512, -1, 0.0, GenTaps, 1234ULL},
  {1024, 1024, -1, 0.0, GenTaps, 1234ULL},
  {1024, 512 + 1, -1, 0.0, GenTaps, 1234ULL},
  {1024, 1024 - 1, -1, 0.0, GenTaps, 1234ULL},
  {1024, 512 + 2, -1, 0.0, GenTaps, 1234ULL},
  {1024, 1024 - 2, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 1, 512, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 1, 1024, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 1, 512 + 1, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 1, 1024 + 1, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 1, 512 + 2, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 1, 1024 - 2, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 2, 512, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 2, 1024, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 2, 512 + 1, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 2, 1024 + 1, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 2, 512 + 2, -1, 0.0, GenTaps, 1234ULL},
  {1024 + 2, 1024 + 2, -1, 0.0, GenTaps, 1234ULL},
  {1024, 512, 10, 100000.0, GenTaps, 1234ULL},

  {1024, 512, -1, 0.0, GenKiss99, 1234ULL},
  {1024, 1024, -1, 0.0, GenKiss99, 1234ULL},
  {1024, 512 + 1, -1, 0.0, GenKiss99, 1234ULL},
  {1024, 1024 - 1, -1, 0.0, GenKiss99, 1234ULL},
  {1024, 512 + 2, -1, 0.0, GenKiss99, 1234ULL},
  {1024, 1024 - 2, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 1, 512, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 1, 1024, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 1, 512 + 1, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 1, 1024 + 1, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 1, 512 + 2, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 1, 1024 - 2, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 2, 512, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 2, 1024, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 2, 512 + 1, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 2, 1024 + 1, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 2, 512 + 2, -1, 0.0, GenKiss99, 1234ULL},
  {1024 + 2, 1024 + 2, -1, 0.0, GenKiss99, 1234ULL},
  {1024, 512, 10, 100000.0, GenKiss99, 1234ULL},
};

TEST_P(SWoRTestD, Result) {
  std::set<int> occurence;
  for (int i = 0; i < params.sampledLen; ++i) {
    auto val = h_outIdx[i];
    // indices must be in the given range
    ASSERT_TRUE(0 <= val && val < params.len)
      << "out-of-range index @i=" << i << " val=" << val
      << " sampledLen=" << params.sampledLen;
    // indices should not repeat
    ASSERT_TRUE(occurence.find(val) == occurence.end())
      << "repeated index @i=" << i << " idx=" << val;
    occurence.insert(val);
  }
  // if there's a skewed distribution, the top index should correspond to the
  // particular item with a large weight
  if (params.largeWeightIndex >= 0) {
    ASSERT_EQ(h_outIdx[0], params.largeWeightIndex);
  }
}
INSTANTIATE_TEST_CASE_P(SWoRTests, SWoRTestD, ::testing::ValuesIn(inputsd));

}  // end namespace Random
}  // end namespace MLCommon
