/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/eltwise.h"
#include "random/rng.h"
#include "stats/sum.h"
#include "test_utils.h"

namespace MLCommon {
namespace Stats {

template <typename T>
struct SumInputs {
  T tolerance;
  int rows, cols;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const SumInputs<T> &dims) {
  return os;
}

template <typename T>
class SumTest : public ::testing::TestWithParam<SumInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<SumInputs<T>>::GetParam();
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(data, len);

    T data_h[len];
    for (int i = 0; i < len; i++) {
      data_h[i] = T(1);
    }

    updateDevice(data, data_h, len, stream);

    allocate(sum_act, cols);
    sum(sum_act, data, cols, rows, false, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(sum_act));
  }

 protected:
  SumInputs<T> params;
  T *data, *sum_act;
};

const std::vector<SumInputs<float>> inputsf = {{0.05f, 1024, 32, 1234ULL},
                                               {0.05f, 1024, 256, 1234ULL}};

const std::vector<SumInputs<double>> inputsd = {{0.05, 1024, 32, 1234ULL},
                                                {0.05, 1024, 256, 1234ULL}};

typedef SumTest<float> SumTestF;
TEST_P(SumTestF, Result) {
  ASSERT_TRUE(devArrMatch(float(params.rows), sum_act, params.cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef SumTest<double> SumTestD;
TEST_P(SumTestD, Result) {
  ASSERT_TRUE(devArrMatch(double(params.rows), sum_act, params.cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(SumTests, SumTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(SumTests, SumTestD, ::testing::ValuesIn(inputsd));

}  // end namespace Stats
}  // end namespace MLCommon
