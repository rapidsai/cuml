#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/distance_type.h>
#include <algorithm>
#include <cuml/common/cuml_allocator.hpp>
#include <iostream>
#include <metrics/batched/silhouette_score.cuh>
#include <metrics/silhouette_score.cuh>
#include <random>
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

//parameter structure definition
struct silhouetteScoreParam {
  int nRows;
  int nCols;
  int nLabels;
  raft::distance::DistanceType metric;
  int chunk;
  double tolerance;
};

//test fixture class
template <typename LabelT, typename DataT>
class silhouetteScoreTest
  : public ::testing::TestWithParam<silhouetteScoreParam> {
 protected:
  void host_silhouette_score() {
    //generating random value test input
    std::vector<double> h_X(nElements, 0.0);
    std::vector<int> h_labels(nRows, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_int_distribution<int> intGenerator(0, nLabels - 1);
    std::uniform_real_distribution<double> realGenerator(0, 100);

    std::generate(h_X.begin(), h_X.end(), [&]() { return realGenerator(dre); });
    std::generate(h_labels.begin(), h_labels.end(),
                  [&]() { return intGenerator(dre); });

    //allocating and initializing memory to the GPU
    CUDA_CHECK(hipStreamCreate(&stream));
    raft::allocate(d_X, nElements, true);
    raft::allocate(d_labels, nElements, true);
    raft::allocate(sampleSilScore, nElements);

    raft::update_device(d_X, &h_X[0], (int)nElements, stream);
    raft::update_device(d_labels, &h_labels[0], (int)nElements, stream);

    //finding the distance matrix

    device_buffer<double> d_distanceMatrix(allocator, stream, nRows * nRows);
    device_buffer<char> workspace(allocator, stream, 1);
    double *h_distanceMatrix =
      (double *)malloc(nRows * nRows * sizeof(double *));

    MLCommon::Distance::pairwise_distance(d_X, d_X, d_distanceMatrix.data(),
                                          nRows, nRows, nCols, workspace,
                                          params.metric, stream);

    CUDA_CHECK(hipStreamSynchronize(stream));

    raft::update_host(h_distanceMatrix, d_distanceMatrix.data(), nRows * nRows,
                      stream);

    //finding the bincount array

    double *binCountArray = (double *)malloc(nLabels * sizeof(double *));
    memset(binCountArray, 0, nLabels * sizeof(double));

    for (int i = 0; i < nRows; ++i) {
      binCountArray[h_labels[i]] += 1;
    }

    //finding the average intra cluster distance for every element

    double *a = (double *)malloc(nRows * sizeof(double *));

    for (int i = 0; i < nRows; ++i) {
      int myLabel = h_labels[i];
      double sumOfIntraClusterD = 0;

      for (int j = 0; j < nRows; ++j) {
        if (h_labels[j] == myLabel) {
          sumOfIntraClusterD += h_distanceMatrix[i * nRows + j];
        }
      }

      if (binCountArray[myLabel] <= 1)
        a[i] = -1;
      else
        a[i] = sumOfIntraClusterD / (binCountArray[myLabel] - 1);
    }

    //finding the average inter cluster distance for every element

    double *b = (double *)malloc(nRows * sizeof(double *));

    for (int i = 0; i < nRows; ++i) {
      int myLabel = h_labels[i];
      double minAvgInterCD = ULLONG_MAX;

      for (int j = 0; j < nLabels; ++j) {
        int curClLabel = j;
        if (curClLabel == myLabel) continue;
        double avgInterCD = 0;

        for (int k = 0; k < nRows; ++k) {
          if (h_labels[k] == curClLabel) {
            avgInterCD += h_distanceMatrix[i * nRows + k];
          }
        }

        if (binCountArray[curClLabel])
          avgInterCD /= binCountArray[curClLabel];
        else
          avgInterCD = ULLONG_MAX;
        minAvgInterCD = min(minAvgInterCD, avgInterCD);
      }

      b[i] = minAvgInterCD;
    }

    //finding the silhouette score for every element

    double *truthSampleSilScore = (double *)malloc(nRows * sizeof(double *));
    for (int i = 0; i < nRows; ++i) {
      if (a[i] == -1)
        truthSampleSilScore[i] = 0;
      else if (a[i] == 0 && b[i] == 0)
        truthSampleSilScore[i] = 0;
      else
        truthSampleSilScore[i] = (b[i] - a[i]) / max(a[i], b[i]);
      truthSilhouetteScore += truthSampleSilScore[i];
    }

    truthSilhouetteScore /= nRows;
  }

  //the constructor
  void SetUp() override {
    //getting the parameters
    params = ::testing::TestWithParam<silhouetteScoreParam>::GetParam();

    nRows = params.nRows;
    nCols = params.nCols;
    nLabels = params.nLabels;
    chunk = params.chunk;
    nElements = nRows * nCols;

    allocator = std::make_shared<raft::mr::device::default_allocator>();

    host_silhouette_score();

    //calling the silhouette_score CUDA implementation
    computedSilhouetteScore = MLCommon::Metrics::silhouette_score(
      d_X, nRows, nCols, d_labels, nLabels, sampleSilScore, allocator, stream,
      params.metric);

    batchedSilhouetteScore =
      Batched::silhouette_score(handle, d_X, nRows, nCols, d_labels, nLabels,
                                sampleSilScore, chunk, params.metric);
  }

  //the destructor
  void TearDown() override {
    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_labels));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  //declaring the data values
  silhouetteScoreParam params;
  int nLabels;
  DataT *d_X = nullptr;
  DataT *sampleSilScore = nullptr;
  LabelT *d_labels = nullptr;
  int nRows;
  int nCols;
  int nElements;
  double truthSilhouetteScore = 0;
  double computedSilhouetteScore = 0;
  double batchedSilhouetteScore = 0;
  hipStream_t stream;
  raft::handle_t handle;
  int chunk;
  std::shared_ptr<MLCommon::deviceAllocator> allocator;
};

//setting test parameter values
const std::vector<silhouetteScoreParam> inputs = {
  {4, 2, 3, raft::distance::DistanceType::L2Expanded, 4, 0.00001},
  {4, 2, 2, raft::distance::DistanceType::L2SqrtUnexpanded, 2, 0.00001},
  {8, 8, 3, raft::distance::DistanceType::L2Unexpanded, 4, 0.00001},
  {11, 2, 5, raft::distance::DistanceType::L2Expanded, 3, 0.00001},
  {40, 2, 8, raft::distance::DistanceType::L2Expanded, 10, 0.00001},
  {12, 7, 3, raft::distance::DistanceType::CosineExpanded, 8, 0.00001},
  {7, 5, 5, raft::distance::DistanceType::L1, 2, 0.00001}};

//writing the test suite
typedef silhouetteScoreTest<int, double> silhouetteScoreTestClass;
TEST_P(silhouetteScoreTestClass, Result) {
  ASSERT_NEAR(computedSilhouetteScore, truthSilhouetteScore, params.tolerance);
  ASSERT_NEAR(batchedSilhouetteScore, truthSilhouetteScore, params.tolerance);
}
INSTANTIATE_TEST_CASE_P(silhouetteScore, silhouetteScoreTestClass,
                        ::testing::ValuesIn(inputs));

}  //end namespace Metrics
}  //end namespace MLCommon
