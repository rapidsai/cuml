/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include <functions/logisticReg.cuh>
#include <raft/random/rng.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Functions {

template <typename T>
struct LogRegLossInputs {
  T tolerance;
  T n_rows;
  T n_cols;
  int len;
};

template <typename T>
class LogRegLossTest : public ::testing::TestWithParam<LogRegLossInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<LogRegLossInputs<T>>::GetParam();
    int len = params.len;
    int n_rows = params.n_rows;
    int n_cols = params.n_cols;

    T *labels, *coef;

    raft::handle_t handle;

    hipStream_t stream = handle.get_stream();

    allocator.reset(new raft::mr::device::default_allocator);

    raft::allocate(in, len);
    raft::allocate(out, 1);
    raft::allocate(out_lasso, 1);
    raft::allocate(out_ridge, 1);
    raft::allocate(out_elasticnet, 1);
    raft::allocate(out_grad, n_cols);
    raft::allocate(out_lasso_grad, n_cols);
    raft::allocate(out_ridge_grad, n_cols);
    raft::allocate(out_elasticnet_grad, n_cols);
    raft::allocate(out_ref, 1);
    raft::allocate(out_lasso_ref, 1);
    raft::allocate(out_ridge_ref, 1);
    raft::allocate(out_elasticnet_ref, 1);
    raft::allocate(out_grad_ref, n_cols);
    raft::allocate(out_lasso_grad_ref, n_cols);
    raft::allocate(out_ridge_grad_ref, n_cols);
    raft::allocate(out_elasticnet_grad_ref, n_cols);

    raft::allocate(labels, params.n_rows);
    raft::allocate(coef, params.n_cols);

    T h_in[len] = {0.1, 0.35, -0.9, -1.4, 2.0, 3.1};
    raft::update_device(in, h_in, len, stream);

    T h_labels[n_rows] = {0.3, 2.0, -1.1};
    raft::update_device(labels, h_labels, n_rows, stream);

    T h_coef[n_cols] = {0.35, -0.24};
    raft::update_device(coef, h_coef, n_cols, stream);

    T h_out_ref[1] = {0.38752545};
    raft::update_device(out_ref, h_out_ref, 1, stream);

    T h_out_lasso_ref[1] = {0.74152};
    raft::update_device(out_lasso_ref, h_out_lasso_ref, 1, stream);

    T h_out_ridge_ref[1] = {0.4955854};
    raft::update_device(out_ridge_ref, h_out_ridge_ref, 1, stream);

    T h_out_elasticnet_ref[1] = {0.618555};
    raft::update_device(out_elasticnet_ref, h_out_elasticnet_ref, 1, stream);

    T h_out_grad_ref[n_cols] = {-0.58284, 0.207666};
    raft::update_device(out_grad_ref, h_out_grad_ref, n_cols, stream);

    T h_out_lasso_grad_ref[n_cols] = {0.0171, -0.39233};
    raft::update_device(out_lasso_grad_ref, h_out_lasso_grad_ref, n_cols,
                        stream);

    T h_out_ridge_grad_ref[n_cols] = {-0.16284, -0.080333};
    raft::update_device(out_ridge_grad_ref, h_out_ridge_grad_ref, n_cols,
                        stream);

    T h_out_elasticnet_grad_ref[n_cols] = {-0.07284, -0.23633};
    raft::update_device(out_elasticnet_grad_ref, h_out_elasticnet_grad_ref,
                        n_cols, stream);

    T alpha = 0.6;
    T l1_ratio = 0.5;

    logisticRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef, out,
                    penalty::NONE, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    logisticRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                         out_grad, penalty::NONE, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    logisticRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef,
                    out_lasso, penalty::L1, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    logisticRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                         out_lasso_grad, penalty::L1, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    logisticRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef,
                    out_ridge, penalty::L2, alpha, l1_ratio, stream);

    logisticRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                         out_ridge_grad, penalty::L2, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    logisticRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef,
                    out_elasticnet, penalty::ELASTICNET, alpha, l1_ratio,
                    stream);

    logisticRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                         out_elasticnet_grad, penalty::ELASTICNET, alpha,
                         l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(coef));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_lasso));
    CUDA_CHECK(hipFree(out_ridge));
    CUDA_CHECK(hipFree(out_elasticnet));
    CUDA_CHECK(hipFree(out_grad));
    CUDA_CHECK(hipFree(out_lasso_grad));
    CUDA_CHECK(hipFree(out_ridge_grad));
    CUDA_CHECK(hipFree(out_elasticnet_grad));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out_lasso_ref));
    CUDA_CHECK(hipFree(out_ridge_ref));
    CUDA_CHECK(hipFree(out_elasticnet_ref));
    CUDA_CHECK(hipFree(out_grad_ref));
    CUDA_CHECK(hipFree(out_lasso_grad_ref));
    CUDA_CHECK(hipFree(out_ridge_grad_ref));
    CUDA_CHECK(hipFree(out_elasticnet_grad_ref));
  }

 protected:
  LogRegLossInputs<T> params;
  T *in;
  T *out, *out_lasso, *out_ridge, *out_elasticnet;
  T *out_ref, *out_lasso_ref, *out_ridge_ref, *out_elasticnet_ref;
  T *out_grad, *out_lasso_grad, *out_ridge_grad, *out_elasticnet_grad;
  T *out_grad_ref, *out_lasso_grad_ref, *out_ridge_grad_ref,
    *out_elasticnet_grad_ref;
  std::shared_ptr<deviceAllocator> allocator;
};

const std::vector<LogRegLossInputs<float>> inputsf = {{0.01f, 3, 2, 6}};

const std::vector<LogRegLossInputs<double>> inputsd = {{0.01, 3, 2, 6}};

typedef LogRegLossTest<float> LogRegLossTestF;
TEST_P(LogRegLossTestF, Result) {
  ASSERT_TRUE(raft::devArrMatch(out_ref, out, 1,
                                raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_lasso_ref, out_lasso, 1,
                                raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_ridge_ref, out_ridge, 1,
                                raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                                raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_grad_ref, out_grad, params.n_cols,
                                raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_lasso_grad_ref, out_lasso_grad,
                                params.n_cols,
                                raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_ridge_grad_ref, out_ridge_grad,
                                params.n_cols,
                                raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                                params.n_cols,
                                raft::CompareApprox<float>(params.tolerance)));
}

typedef LogRegLossTest<double> LogRegLossTestD;
TEST_P(LogRegLossTestD, Result) {
  ASSERT_TRUE(raft::devArrMatch(out_ref, out, 1,
                                raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_lasso_ref, out_lasso, 1,
                                raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_ridge_ref, out_ridge, 1,
                                raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                                raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_grad_ref, out_grad, params.n_cols,
                                raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_lasso_grad_ref, out_lasso_grad,
                                params.n_cols,
                                raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_ridge_grad_ref, out_ridge_grad,
                                params.n_cols,
                                raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                                params.n_cols,
                                raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(LogRegLossTests, LogRegLossTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(LogRegLossTests, LogRegLossTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Functions
}  // end namespace MLCommon
