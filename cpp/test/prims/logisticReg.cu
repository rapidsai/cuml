#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "functions/logisticReg.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace Functions {

template <typename T>
struct LogRegLossInputs {
  T tolerance;
  T n_rows;
  T n_cols;
  int len;
};

template <typename T>
class LogRegLossTest : public ::testing::TestWithParam<LogRegLossInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<LogRegLossInputs<T>>::GetParam();
    int len = params.len;
    int n_rows = params.n_rows;
    int n_cols = params.n_cols;

    T *labels, *coef;

    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    allocator.reset(new defaultDeviceAllocator);

    allocate(in, len);
    allocate(out, 1);
    allocate(out_lasso, 1);
    allocate(out_ridge, 1);
    allocate(out_elasticnet, 1);
    allocate(out_grad, n_cols);
    allocate(out_lasso_grad, n_cols);
    allocate(out_ridge_grad, n_cols);
    allocate(out_elasticnet_grad, n_cols);
    allocate(out_ref, 1);
    allocate(out_lasso_ref, 1);
    allocate(out_ridge_ref, 1);
    allocate(out_elasticnet_ref, 1);
    allocate(out_grad_ref, n_cols);
    allocate(out_lasso_grad_ref, n_cols);
    allocate(out_ridge_grad_ref, n_cols);
    allocate(out_elasticnet_grad_ref, n_cols);

    allocate(labels, params.n_rows);
    allocate(coef, params.n_cols);

    T h_in[len] = {0.1, 0.35, -0.9, -1.4, 2.0, 3.1};
    updateDevice(in, h_in, len, stream);

    T h_labels[n_rows] = {0.3, 2.0, -1.1};
    updateDevice(labels, h_labels, n_rows, stream);

    T h_coef[n_cols] = {0.35, -0.24};
    updateDevice(coef, h_coef, n_cols, stream);

    T h_out_ref[1] = {0.38752545};
    updateDevice(out_ref, h_out_ref, 1, stream);

    T h_out_lasso_ref[1] = {0.74152};
    updateDevice(out_lasso_ref, h_out_lasso_ref, 1, stream);

    T h_out_ridge_ref[1] = {0.4955854};
    updateDevice(out_ridge_ref, h_out_ridge_ref, 1, stream);

    T h_out_elasticnet_ref[1] = {0.618555};
    updateDevice(out_elasticnet_ref, h_out_elasticnet_ref, 1, stream);

    T h_out_grad_ref[n_cols] = {-0.58284, 0.207666};
    updateDevice(out_grad_ref, h_out_grad_ref, n_cols, stream);

    T h_out_lasso_grad_ref[n_cols] = {0.0171, -0.39233};
    updateDevice(out_lasso_grad_ref, h_out_lasso_grad_ref, n_cols, stream);

    T h_out_ridge_grad_ref[n_cols] = {-0.16284, -0.080333};
    updateDevice(out_ridge_grad_ref, h_out_ridge_grad_ref, n_cols, stream);

    T h_out_elasticnet_grad_ref[n_cols] = {-0.07284, -0.23633};
    updateDevice(out_elasticnet_grad_ref, h_out_elasticnet_grad_ref, n_cols,
                 stream);

    T alpha = 0.6;
    T l1_ratio = 0.5;

    logisticRegLoss(in, params.n_rows, params.n_cols, labels, coef, out,
                    penalty::NONE, alpha, l1_ratio, cublas_handle, allocator,
                    stream);

    updateDevice(in, h_in, len, stream);

    logisticRegLossGrads(in, params.n_rows, params.n_cols, labels, coef,
                         out_grad, penalty::NONE, alpha, l1_ratio,
                         cublas_handle, allocator, stream);

    updateDevice(in, h_in, len, stream);

    logisticRegLoss(in, params.n_rows, params.n_cols, labels, coef, out_lasso,
                    penalty::L1, alpha, l1_ratio, cublas_handle, allocator,
                    stream);

    updateDevice(in, h_in, len, stream);

    logisticRegLossGrads(in, params.n_rows, params.n_cols, labels, coef,
                         out_lasso_grad, penalty::L1, alpha, l1_ratio,
                         cublas_handle, allocator, stream);

    updateDevice(in, h_in, len, stream);

    logisticRegLoss(in, params.n_rows, params.n_cols, labels, coef, out_ridge,
                    penalty::L2, alpha, l1_ratio, cublas_handle, allocator,
                    stream);

    logisticRegLossGrads(in, params.n_rows, params.n_cols, labels, coef,
                         out_ridge_grad, penalty::L2, alpha, l1_ratio,
                         cublas_handle, allocator, stream);

    updateDevice(in, h_in, len, stream);

    logisticRegLoss(in, params.n_rows, params.n_cols, labels, coef,
                    out_elasticnet, penalty::ELASTICNET, alpha, l1_ratio,
                    cublas_handle, allocator, stream);

    logisticRegLossGrads(in, params.n_rows, params.n_cols, labels, coef,
                         out_elasticnet_grad, penalty::ELASTICNET, alpha,
                         l1_ratio, cublas_handle, allocator, stream);

    updateDevice(in, h_in, len, stream);

    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(coef));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_lasso));
    CUDA_CHECK(hipFree(out_ridge));
    CUDA_CHECK(hipFree(out_elasticnet));
    CUDA_CHECK(hipFree(out_grad));
    CUDA_CHECK(hipFree(out_lasso_grad));
    CUDA_CHECK(hipFree(out_ridge_grad));
    CUDA_CHECK(hipFree(out_elasticnet_grad));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out_lasso_ref));
    CUDA_CHECK(hipFree(out_ridge_ref));
    CUDA_CHECK(hipFree(out_elasticnet_ref));
    CUDA_CHECK(hipFree(out_grad_ref));
    CUDA_CHECK(hipFree(out_lasso_grad_ref));
    CUDA_CHECK(hipFree(out_ridge_grad_ref));
    CUDA_CHECK(hipFree(out_elasticnet_grad_ref));
  }

 protected:
  LogRegLossInputs<T> params;
  T *in;
  T *out, *out_lasso, *out_ridge, *out_elasticnet;
  T *out_ref, *out_lasso_ref, *out_ridge_ref, *out_elasticnet_ref;
  T *out_grad, *out_lasso_grad, *out_ridge_grad, *out_elasticnet_grad;
  T *out_grad_ref, *out_lasso_grad_ref, *out_ridge_grad_ref,
    *out_elasticnet_grad_ref;
  std::shared_ptr<deviceAllocator> allocator;
};

const std::vector<LogRegLossInputs<float>> inputsf = {{0.01f, 3, 2, 6}};

const std::vector<LogRegLossInputs<double>> inputsd = {{0.01, 3, 2, 6}};

typedef LogRegLossTest<float> LogRegLossTestF;
TEST_P(LogRegLossTestF, Result) {
  ASSERT_TRUE(
    devArrMatch(out_ref, out, 1, CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_grad_ref, out_grad, params.n_cols,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.n_cols,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.n_cols,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.n_cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef LogRegLossTest<double> LogRegLossTestD;
TEST_P(LogRegLossTestD, Result) {
  ASSERT_TRUE(
    devArrMatch(out_ref, out, 1, CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_grad_ref, out_grad, params.n_cols,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.n_cols,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.n_cols,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.n_cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(LogRegLossTests, LogRegLossTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(LogRegLossTests, LogRegLossTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Functions
}  // end namespace MLCommon
