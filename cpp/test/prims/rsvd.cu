#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <cuda_utils.cuh>
#include <linalg/rsvd.cuh>
#include <random/rng.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct RsvdInputs {
  T tolerance;
  int n_row;
  int n_col;
  T PC_perc;
  T UpS_perc;
  int k;
  int p;
  bool use_bbt;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const RsvdInputs<T> &dims) {
  return os;
}

template <typename T>
class RsvdTest : public ::testing::TestWithParam<RsvdInputs<T>> {
 protected:
  void SetUp() override {
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreate(&stream));
    allocator.reset(new raft::mr::device::default_allocator);

    params = ::testing::TestWithParam<RsvdInputs<T>>::GetParam();
    // rSVD seems to be very sensitive to the random number sequence as well!
    Random::Rng r(params.seed, Random::GenTaps);
    int m = params.n_row, n = params.n_col;
    T eig_svd_tol = 1.e-7;
    int max_sweeps = 100;

    T mu = 0.0, sigma = 1.0;
    allocate(A, m * n);
    if (params.tolerance > 1) {  // Sanity check
      ASSERT(m == 3, "This test only supports mxn=3x2!");
      ASSERT(m * n == 6, "This test only supports mxn=3x2!");
      T data_h[] = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
      updateDevice(A, data_h, m * n, stream);

      T left_eig_vectors_ref_h[] = {-0.308219, -0.906133, -0.289695};
      T right_eig_vectors_ref_h[] = {-0.638636, -0.769509};
      T sing_vals_ref_h[] = {7.065283};

      allocate(left_eig_vectors_ref, m * 1);
      allocate(right_eig_vectors_ref, n * 1);
      allocate(sing_vals_ref, 1);

      updateDevice(left_eig_vectors_ref, left_eig_vectors_ref_h, m * 1, stream);
      updateDevice(right_eig_vectors_ref, right_eig_vectors_ref_h, n * 1,
                   stream);
      updateDevice(sing_vals_ref, sing_vals_ref_h, 1, stream);

    } else {  // Other normal tests
      r.normal(A, m * n, mu, sigma, stream);
    }
    A_backup_cpu = (T *)malloc(
      sizeof(T) * m *
      n);  // Backup A matrix as svdJacobi will destroy the content of A
    updateHost(A_backup_cpu, A, m * n, stream);

    // RSVD tests
    if (params.k == 0) {  // Test with PC and upsampling ratio
      params.k = max((int)(min(m, n) * params.PC_perc), 1);
      params.p = max((int)(min(m, n) * params.UpS_perc), 1);
      allocate(U, m * params.k, true);
      allocate(S, params.k, true);
      allocate(V, n * params.k, true);
      rsvdPerc(A, m, n, S, U, V, params.PC_perc, params.UpS_perc,
               params.use_bbt, true, true, false, eig_svd_tol, max_sweeps,
               cusolverH, cublasH, stream, allocator);
    } else {  // Test with directly given fixed rank
      allocate(U, m * params.k, true);
      allocate(S, params.k, true);
      allocate(V, n * params.k, true);
      rsvdFixedRank(A, m, n, S, U, V, params.k, params.p, params.use_bbt, true,
                    true, true, eig_svd_tol, max_sweeps, cusolverH, cublasH,
                    stream, allocator);
    }
    updateDevice(A, A_backup_cpu, m * n, stream);

    free(A_backup_cpu);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(A));
    CUDA_CHECK(hipFree(U));
    CUDA_CHECK(hipFree(S));
    CUDA_CHECK(hipFree(V));
    if (left_eig_vectors_ref) CUDA_CHECK(hipFree(left_eig_vectors_ref));
    if (right_eig_vectors_ref) CUDA_CHECK(hipFree(right_eig_vectors_ref));
    if (sing_vals_ref) CUDA_CHECK(hipFree(sing_vals_ref));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  RsvdInputs<T> params;
  T *A, *A_backup_cpu,
    *U = nullptr, *S = nullptr, *V = nullptr, *left_eig_vectors_ref = nullptr,
    *right_eig_vectors_ref = nullptr, *sing_vals_ref = nullptr;
  hipsolverHandle_t cusolverH = nullptr;
  hipblasHandle_t cublasH = nullptr;
  hipStream_t stream;
  std::shared_ptr<deviceAllocator> allocator;
};

const std::vector<RsvdInputs<float>> inputs_fx = {
  // Test with ratios
  {0.20f, 256, 256, 0.2f, 0.05f, 0, 0, true, 4321ULL},     // Square + BBT
  {0.20f, 2048, 256, 0.2f, 0.05f, 0, 0, true, 4321ULL},    // Tall + BBT
  {0.20f, 256, 256, 0.2f, 0.05f, 0, 0, false, 4321ULL},    // Square + non-BBT
  {0.20f, 2048, 256, 0.2f, 0.05f, 0, 0, false, 4321ULL},   // Tall + non-BBT
  {0.20f, 2048, 2048, 0.2f, 0.05f, 0, 0, true, 4321ULL},   // Square + BBT
  {0.60f, 16384, 2048, 0.2f, 0.05f, 0, 0, true, 4321ULL},  // Tall + BBT
  {0.20f, 2048, 2048, 0.2f, 0.05f, 0, 0, false, 4321ULL},  // Square + non-BBT
  {0.60f, 16384, 2048, 0.2f, 0.05f, 0, 0, false, 4321ULL}  // Tall + non-BBT

  ,  // Test with fixed ranks
  {0.10f, 256, 256, 0.0f, 0.0f, 100, 5, true, 4321ULL},     // Square + BBT
  {0.12f, 2048, 256, 0.0f, 0.0f, 100, 5, true, 4321ULL},    // Tall + BBT
  {0.10f, 256, 256, 0.0f, 0.0f, 100, 5, false, 4321ULL},    // Square + non-BBT
  {0.12f, 2048, 256, 0.0f, 0.0f, 100, 5, false, 4321ULL},   // Tall + non-BBT
  {0.60f, 2048, 2048, 0.0f, 0.0f, 100, 5, true, 4321ULL},   // Square + BBT
  {1.00f, 16384, 2048, 0.0f, 0.0f, 100, 5, true, 4321ULL},  // Tall + BBT
  {0.60f, 2048, 2048, 0.0f, 0.0f, 100, 5, false, 4321ULL},  // Square + non-BBT
  {1.00f, 16384, 2048, 0.0f, 0.0f, 100, 5, false, 4321ULL}  // Tall + non-BBT
};

const std::vector<RsvdInputs<double>> inputs_dx = {
  // Test with ratios
  {0.20, 256, 256, 0.2, 0.05, 0, 0, true, 4321ULL},     // Square + BBT
  {0.20, 2048, 256, 0.2, 0.05, 0, 0, true, 4321ULL},    // Tall + BBT
  {0.20, 256, 256, 0.2, 0.05, 0, 0, false, 4321ULL},    // Square + non-BBT
  {0.20, 2048, 256, 0.2, 0.05, 0, 0, false, 4321ULL},   // Tall + non-BBT
  {0.20, 2048, 2048, 0.2, 0.05, 0, 0, true, 4321ULL},   // Square + BBT
  {0.60, 16384, 2048, 0.2, 0.05, 0, 0, true, 4321ULL},  // Tall + BBT
  {0.20, 2048, 2048, 0.2, 0.05, 0, 0, false, 4321ULL},  // Square + non-BBT
  {0.60, 16384, 2048, 0.2, 0.05, 0, 0, false, 4321ULL}  // Tall + non-BBT

  ,                                                     // Test with fixed ranks
  {0.10, 256, 256, 0.0, 0.0, 100, 5, true, 4321ULL},    // Square + BBT
  {0.12, 2048, 256, 0.0, 0.0, 100, 5, true, 4321ULL},   // Tall + BBT
  {0.10, 256, 256, 0.0, 0.0, 100, 5, false, 4321ULL},   // Square + non-BBT
  {0.12, 2048, 256, 0.0, 0.0, 100, 5, false, 4321ULL},  // Tall + non-BBT
  {0.60, 2048, 2048, 0.0, 0.0, 100, 5, true, 4321ULL},  // Square + BBT
  {1.00, 16384, 2048, 0.0, 0.0, 100, 5, true, 4321ULL},  // Tall + BBT
  {0.60, 2048, 2048, 0.0, 0.0, 100, 5, false, 4321ULL},  // Square + non-BBT
  {1.00, 16384, 2048, 0.0, 0.0, 100, 5, false, 4321ULL}  // Tall + non-BBT
};

const std::vector<RsvdInputs<float>> sanity_inputs_fx = {
  {100000000000000000.0f, 3, 2, 0.2f, 0.05f, 0, 0, true, 4321ULL},
  {100000000000000000.0f, 3, 2, 0.0f, 0.0f, 1, 1, true, 4321ULL},
  {100000000000000000.0f, 3, 2, 0.2f, 0.05f, 0, 0, false, 4321ULL},
  {100000000000000000.0f, 3, 2, 0.0f, 0.0f, 1, 1, false, 4321ULL}};

const std::vector<RsvdInputs<double>> sanity_inputs_dx = {
  {100000000000000000.0, 3, 2, 0.2, 0.05, 0, 0, true, 4321ULL},
  {100000000000000000.0, 3, 2, 0.0, 0.0, 1, 1, true, 4321ULL},
  {100000000000000000.0, 3, 2, 0.2, 0.05, 0, 0, false, 4321ULL},
  {100000000000000000.0, 3, 2, 0.0, 0.0, 1, 1, false, 4321ULL}};

typedef RsvdTest<float> RsvdSanityCheckValF;
TEST_P(RsvdSanityCheckValF, Result) {
  ASSERT_TRUE(devArrMatch(sing_vals_ref, S, params.k,
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef RsvdTest<double> RsvdSanityCheckValD;
TEST_P(RsvdSanityCheckValD, Result) {
  ASSERT_TRUE(devArrMatch(sing_vals_ref, S, params.k,
                          CompareApproxAbs<double>(params.tolerance)));
}

typedef RsvdTest<float> RsvdSanityCheckLeftVecF;
TEST_P(RsvdSanityCheckLeftVecF, Result) {
  ASSERT_TRUE(devArrMatch(left_eig_vectors_ref, U, params.n_row * params.k,
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef RsvdTest<double> RsvdSanityCheckLeftVecD;
TEST_P(RsvdSanityCheckLeftVecD, Result) {
  ASSERT_TRUE(devArrMatch(left_eig_vectors_ref, U, params.n_row * params.k,
                          CompareApproxAbs<double>(params.tolerance)));
}

typedef RsvdTest<float> RsvdSanityCheckRightVecF;
TEST_P(RsvdSanityCheckRightVecF, Result) {
  ASSERT_TRUE(devArrMatch(right_eig_vectors_ref, V, params.n_col * params.k,
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef RsvdTest<double> RsvdSanityCheckRightVecD;
TEST_P(RsvdSanityCheckRightVecD, Result) {
  ASSERT_TRUE(devArrMatch(right_eig_vectors_ref, V, params.n_col * params.k,
                          CompareApproxAbs<double>(params.tolerance)));
}

typedef RsvdTest<float> RsvdTestSquareMatrixNormF;
TEST_P(RsvdTestSquareMatrixNormF, Result) {
  hipblasHandle_t cublasH;
  CUBLAS_CHECK(hipblasCreate(&cublasH));
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  std::shared_ptr<deviceAllocator> allocator(
    new raft::mr::device::default_allocator);
  ASSERT_TRUE(evaluateSVDByL2Norm(A, U, S, V, params.n_row, params.n_col,
                                  params.k, 4 * params.tolerance, cublasH,
                                  stream, allocator));
  CUBLAS_CHECK(hipblasDestroy(cublasH));
  CUDA_CHECK(hipStreamDestroy(stream));
}

typedef RsvdTest<double> RsvdTestSquareMatrixNormD;
TEST_P(RsvdTestSquareMatrixNormD, Result) {
  hipblasHandle_t cublasH;
  CUBLAS_CHECK(hipblasCreate(&cublasH));
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  std::shared_ptr<deviceAllocator> allocator(
    new raft::mr::device::default_allocator);
  ASSERT_TRUE(evaluateSVDByL2Norm(A, U, S, V, params.n_row, params.n_col,
                                  params.k, 4 * params.tolerance, cublasH,
                                  stream, allocator));
  CUBLAS_CHECK(hipblasDestroy(cublasH));
  CUDA_CHECK(hipStreamDestroy(stream));
}

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdSanityCheckValF,
                        ::testing::ValuesIn(sanity_inputs_fx));

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdSanityCheckValD,
                        ::testing::ValuesIn(sanity_inputs_dx));

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdSanityCheckLeftVecF,
                        ::testing::ValuesIn(sanity_inputs_fx));

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdSanityCheckLeftVecD,
                        ::testing::ValuesIn(sanity_inputs_dx));

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdSanityCheckRightVecF,
                        ::testing::ValuesIn(sanity_inputs_fx));

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdSanityCheckRightVecD,
                        ::testing::ValuesIn(sanity_inputs_dx));

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdTestSquareMatrixNormF,
                        ::testing::ValuesIn(inputs_fx));

INSTANTIATE_TEST_CASE_P(RsvdTests, RsvdTestSquareMatrixNormD,
                        ::testing::ValuesIn(inputs_dx));

}  // end namespace LinAlg
}  // end namespace MLCommon
