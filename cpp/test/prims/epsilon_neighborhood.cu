/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <distance/epsilon_neighborhood.cuh>
#include <random/make_blobs.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Distance {

template <typename T, typename IdxT>
struct EpsInputs {
  IdxT n_row, n_col, n_centers, n_batches;
  T eps;
};

template <typename T, typename IdxT>
::std::ostream& operator<<(::std::ostream& os, const EpsInputs<T, IdxT>& p) {
  return os;
}

template <typename T, typename IdxT>
class EpsNeighTest : public ::testing::TestWithParam<EpsInputs<T, IdxT>> {
 protected:
  void SetUp() override {
    param = ::testing::TestWithParam<EpsInputs<T, IdxT>>::GetParam();
    CUDA_CHECK(hipStreamCreate(&stream));
    raft::allocate(data, param.n_row * param.n_col);
    raft::allocate(labels, param.n_row);
    batchSize = param.n_row / param.n_batches;
    raft::allocate(adj, param.n_row * batchSize);
    raft::allocate(vd, batchSize + 1, true);
    allocator.reset(new raft::mr::device::default_allocator);
    Random::make_blobs<T, IdxT>(data, labels, param.n_row, param.n_col,
                                param.n_centers, allocator, stream, true,
                                nullptr, nullptr, T(0.01), false);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(adj));
    CUDA_CHECK(hipFree(vd));
  }

  EpsInputs<T, IdxT> param;
  hipStream_t stream;
  T* data;
  bool* adj;
  IdxT *labels, *vd;
  IdxT batchSize;
  std::shared_ptr<deviceAllocator> allocator;
};  // class EpsNeighTest

const std::vector<EpsInputs<float, int>> inputsfi = {
  {15000, 16, 5, 1, 2.f},     {14000, 16, 5, 1, 2.f},
  {15000, 17, 5, 1, 2.f},     {14000, 17, 5, 1, 2.f},
  {15000, 18, 5, 1, 2.f},     {14000, 18, 5, 1, 2.f},
  {15000, 32, 5, 1, 2.f},     {14000, 32, 5, 1, 2.f},
  {20000, 10000, 10, 1, 2.f}, {20000, 10000, 10, 2, 2.f},
};
typedef EpsNeighTest<float, int> EpsNeighTestFI;
TEST_P(EpsNeighTestFI, Result) {
  for (int i = 0; i < param.n_batches; ++i) {
    CUDA_CHECK(
      hipMemsetAsync(adj, 0, sizeof(bool) * param.n_row * batchSize, stream));
    CUDA_CHECK(hipMemsetAsync(vd, 0, sizeof(int) * (batchSize + 1), stream));
    epsUnexpL2SqNeighborhood<float, int>(
      adj, vd, data, data + (i * batchSize * param.n_col), param.n_row,
      batchSize, param.n_col, param.eps * param.eps, stream);
    ASSERT_TRUE(raft::devArrMatch(param.n_row / param.n_centers, vd, batchSize,
                                  raft::Compare<int>(), stream));
  }
}
INSTANTIATE_TEST_CASE_P(EpsNeighTests, EpsNeighTestFI,
                        ::testing::ValuesIn(inputsfi));

};  // namespace Distance
};  // namespace MLCommon
