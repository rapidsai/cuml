/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "array/array.h"

#include <cuda_utils.h>
#include "test_utils.h"

#include <vector>
#include <iostream>


namespace MLCommon {
namespace Array {


class ArrayTest: public ::testing::Test {

protected:
    void SetUp() override {}
    void TearDown() override {}
};


typedef ArrayTest MakeMonotonicTest;
TEST_F(MakeMonotonicTest, Result) {

    hipStream_t stream;
    CUDA_CHECK( hipStreamCreate(&stream) );

    int m = 12;

    float *data, *actual, *expected;

    allocate(data, m, true);
    allocate(actual, m, true);
    allocate(expected, m, true);

    float* data_h = new float[m]{ 1.0, 2.0, 2.0, 2.0, 2.0, 3.0, 8.0, 7.0, 8.0, 8.0, 25.0, 80.0 };

    float *expected_h = new float[m]{1.0, 2.0, 2.0, 2.0, 2.0, 3.0, 5.0, 4.0, 5.0, 5.0, 6.0, 7.0 };

    updateDevice(data, data_h, m, stream);
    updateDevice(expected, expected_h, m, stream);

    make_monotonic(actual, data, m, stream);

    CUDA_CHECK( hipStreamSynchronize(stream) );

    ASSERT_TRUE(devArrMatch(actual, expected, m, Compare<bool>(), stream));

    CUDA_CHECK( hipStreamDestroy(stream) );
    CUDA_CHECK( hipFree(data) );
    CUDA_CHECK( hipFree(actual) );

    delete data_h;
    delete expected_h;
}
};
};
