#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <linalg/cutlass_gemm.cuh>

namespace MLCommon {
namespace LinAlg {

template <typename T>
__global__ void fillKernel(T *arr, T val, int N) {
  const int stride = blockDim.x * gridDim.x;
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  for (int i = tid; i < N; i += stride) arr[i] = val;
}

template <typename T, int NTHREADS = 256, int NITEMS = 4>
void fill(T *arr, T val, int N) {
  const int nblks = raft::ceildiv<int>(N, NTHREADS * NITEMS);
  fillKernel<T><<<nblks, NTHREADS>>>(arr, val, N);
  CUDA_CHECK(hipPeekAtLastError());
}

TEST(Gemm, Gemm_128x128x8) {
  float *A, *B, *C, *D;
  int M = 128, N = 128, K = 64;
  CUDA_CHECK(hipMalloc((void **)&A, sizeof(float) * M * K));
  fill(A, 1.f, M * K);
  CUDA_CHECK(hipMalloc((void **)&B, sizeof(float) * K * N));
  fill(B, 0.5f, K * N);
  CUDA_CHECK(hipMalloc((void **)&C, sizeof(float) * M * N));
  fill(C, 2.f, M * N);
  CUDA_CHECK(hipMalloc((void **)&D, sizeof(float) * M * N));
  CUDA_CHECK(hipMemset(D, 0, sizeof(float) * M * N));
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  gemm<float, float, float, cutlass::Shape<8, 128, 128>>(
    HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 1.f, B, N, A, K, 1.f, C, N, D, stream);
  float *hD = new float[M * N];
  raft::update_host<float>(hD, D, M * N, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  for (int i = 0; i < M * N; ++i) {
    ASSERT_FLOAT_EQ(0.5f * K + 2.f, hD[i]) << " @hD[" << i << "]";
  }
  delete[] hD;
  CUDA_CHECK(hipStreamDestroy(stream));
  CUDA_CHECK(hipFree(A));
  CUDA_CHECK(hipFree(B));
  CUDA_CHECK(hipFree(C));
  CUDA_CHECK(hipFree(D));
}

}  // namespace LinAlg
}  // namespace MLCommon
