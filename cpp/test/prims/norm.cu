#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/norm.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct NormInputs {
  T tolerance;
  int rows, cols;
  NormType type;
  bool do_sqrt;
  bool rowMajor;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const NormInputs<T> &I) {
  os << "{ " << I.tolerance << ", " << I.rows << ", " << I.cols << ", "
     << I.type << ", " << I.do_sqrt << ", " << I.seed << '}' << std::endl;
  return os;
}

///// Row-wise norm test definitions
template <typename Type>
__global__ void naiveRowNormKernel(Type *dots, const Type *data, int D, int N,
                                   NormType type, bool do_sqrt) {
  Type acc = (Type)0;
  int rowStart = threadIdx.x + blockIdx.x * blockDim.x;
  if (rowStart < N) {
    for (int i = 0; i < D; ++i) {
      if (type == L2Norm) {
        acc += data[rowStart * D + i] * data[rowStart * D + i];
      } else {
        acc += myAbs(data[rowStart * D + i]);
      }
    }
    dots[rowStart] = do_sqrt ? mySqrt(acc) : acc;
  }
}

template <typename Type>
void naiveRowNorm(Type *dots, const Type *data, int D, int N, NormType type,
                  bool do_sqrt, hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(N, TPB);
  naiveRowNormKernel<Type>
    <<<nblks, TPB, 0, stream>>>(dots, data, D, N, type, do_sqrt);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
class RowNormTest : public ::testing::TestWithParam<NormInputs<T>> {
 public:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    params = ::testing::TestWithParam<NormInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols, len = rows * cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(data, len);
    allocate(dots_exp, rows);
    allocate(dots_act, rows);
    r.uniform(data, len, T(-1.0), T(1.0), stream);
    naiveRowNorm(dots_exp, data, cols, rows, params.type, params.do_sqrt,
                 stream);
    if (params.do_sqrt) {
      auto fin_op = [] __device__(T in) { return mySqrt(in); };
      rowNorm(dots_act, data, cols, rows, params.type, params.rowMajor, stream,
              fin_op);
    } else {
      rowNorm(dots_act, data, cols, rows, params.type, params.rowMajor, stream);
    }
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(dots_exp));
    CUDA_CHECK(hipFree(dots_act));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  NormInputs<T> params;
  T *data, *dots_exp, *dots_act;
  hipStream_t stream;
};

///// Column-wise norm test definitisons
template <typename Type>
__global__ void naiveColNormKernel(Type *dots, const Type *data, int D, int N,
                                   NormType type, bool do_sqrt) {
  int colID = threadIdx.x + blockIdx.x * blockDim.x;
  if (colID > D) return;  //avoid out-of-bounds thread

  Type acc = 0;
  for (int i = 0; i < N; i++) {
    Type v = data[colID + i * D];
    acc += type == L2Norm ? v * v : myAbs(v);
  }

  dots[colID] = do_sqrt ? mySqrt(acc) : acc;
}

template <typename Type>
void naiveColNorm(Type *dots, const Type *data, int D, int N, NormType type,
                  bool do_sqrt, hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(D, TPB);
  naiveColNormKernel<Type>
    <<<nblks, TPB, 0, stream>>>(dots, data, D, N, type, do_sqrt);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
class ColNormTest : public ::testing::TestWithParam<NormInputs<T>> {
 public:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    params = ::testing::TestWithParam<NormInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols, len = rows * cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(data, len);
    r.uniform(data, len, T(-1.0), T(1.0), stream);
    allocate(dots_exp, cols);
    allocate(dots_act, cols);

    naiveColNorm(dots_exp, data, cols, rows, params.type, params.do_sqrt,
                 stream);
    if (params.do_sqrt) {
      auto fin_op = [] __device__(T in) { return mySqrt(in); };
      colNorm(dots_act, data, cols, rows, params.type, params.rowMajor, stream,
              fin_op);
    } else {
      colNorm(dots_act, data, cols, rows, params.type, params.rowMajor, stream);
    }
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(dots_exp));
    CUDA_CHECK(hipFree(dots_act));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  NormInputs<T> params;
  T *data, *dots_exp, *dots_act;
  hipStream_t stream;
};

///// Row- and column-wise tests
const std::vector<NormInputs<float>> inputsf = {
  {0.00001f, 1024, 32, L1Norm, false, true, 1234ULL},
  {0.00001f, 1024, 64, L1Norm, false, true, 1234ULL},
  {0.00001f, 1024, 128, L1Norm, false, true, 1234ULL},
  {0.00001f, 1024, 256, L1Norm, false, true, 1234ULL},
  {0.00001f, 1024, 32, L2Norm, false, true, 1234ULL},
  {0.00001f, 1024, 64, L2Norm, false, true, 1234ULL},
  {0.00001f, 1024, 128, L2Norm, false, true, 1234ULL},
  {0.00001f, 1024, 256, L2Norm, false, true, 1234ULL},

  {0.00001f, 1024, 32, L1Norm, true, true, 1234ULL},
  {0.00001f, 1024, 64, L1Norm, true, true, 1234ULL},
  {0.00001f, 1024, 128, L1Norm, true, true, 1234ULL},
  {0.00001f, 1024, 256, L1Norm, true, true, 1234ULL},
  {0.00001f, 1024, 32, L2Norm, true, true, 1234ULL},
  {0.00001f, 1024, 64, L2Norm, true, true, 1234ULL},
  {0.00001f, 1024, 128, L2Norm, true, true, 1234ULL},
  {0.00001f, 1024, 256, L2Norm, true, true, 1234ULL}};

const std::vector<NormInputs<double>> inputsd = {
  {0.00000001, 1024, 32, L1Norm, false, true, 1234ULL},
  {0.00000001, 1024, 64, L1Norm, false, true, 1234ULL},
  {0.00000001, 1024, 128, L1Norm, false, true, 1234ULL},
  {0.00000001, 1024, 256, L1Norm, false, true, 1234ULL},
  {0.00000001, 1024, 32, L2Norm, false, true, 1234ULL},
  {0.00000001, 1024, 64, L2Norm, false, true, 1234ULL},
  {0.00000001, 1024, 128, L2Norm, false, true, 1234ULL},
  {0.00000001, 1024, 256, L2Norm, false, true, 1234ULL},

  {0.00000001, 1024, 32, L1Norm, true, true, 1234ULL},
  {0.00000001, 1024, 64, L1Norm, true, true, 1234ULL},
  {0.00000001, 1024, 128, L1Norm, true, true, 1234ULL},
  {0.00000001, 1024, 256, L1Norm, true, true, 1234ULL},
  {0.00000001, 1024, 32, L2Norm, true, true, 1234ULL},
  {0.00000001, 1024, 64, L2Norm, true, true, 1234ULL},
  {0.00000001, 1024, 128, L2Norm, true, true, 1234ULL},
  {0.00000001, 1024, 256, L2Norm, true, true, 1234ULL}};

typedef RowNormTest<float> RowNormTestF;
TEST_P(RowNormTestF, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.rows,
                          CompareApprox<float>(params.tolerance)));
}

typedef RowNormTest<double> RowNormTestD;
TEST_P(RowNormTestD, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.rows,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(RowNormTests, RowNormTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(RowNormTests, RowNormTestD,
                        ::testing::ValuesIn(inputsd));

const std::vector<NormInputs<float>> inputscf = {
  {0.00001f, 32, 1024, L1Norm, false, true, 1234ULL},
  {0.00001f, 64, 1024, L1Norm, false, true, 1234ULL},
  {0.00001f, 128, 1024, L1Norm, false, true, 1234ULL},
  {0.00001f, 256, 1024, L1Norm, false, true, 1234ULL},
  {0.00001f, 32, 1024, L2Norm, false, true, 1234ULL},
  {0.00001f, 64, 1024, L2Norm, false, true, 1234ULL},
  {0.00001f, 128, 1024, L2Norm, false, true, 1234ULL},
  {0.00001f, 256, 1024, L2Norm, false, true, 1234ULL},

  {0.00001f, 32, 1024, L1Norm, true, true, 1234ULL},
  {0.00001f, 64, 1024, L1Norm, true, true, 1234ULL},
  {0.00001f, 128, 1024, L1Norm, true, true, 1234ULL},
  {0.00001f, 256, 1024, L1Norm, true, true, 1234ULL},
  {0.00001f, 32, 1024, L2Norm, true, true, 1234ULL},
  {0.00001f, 64, 1024, L2Norm, true, true, 1234ULL},
  {0.00001f, 128, 1024, L2Norm, true, true, 1234ULL},
  {0.00001f, 256, 1024, L2Norm, true, true, 1234ULL}};

const std::vector<NormInputs<double>> inputscd = {
  {0.00000001, 32, 1024, L1Norm, false, true, 1234ULL},
  {0.00000001, 64, 1024, L1Norm, false, true, 1234ULL},
  {0.00000001, 128, 1024, L1Norm, false, true, 1234ULL},
  {0.00000001, 256, 1024, L1Norm, false, true, 1234ULL},
  {0.00000001, 32, 1024, L2Norm, false, true, 1234ULL},
  {0.00000001, 64, 1024, L2Norm, false, true, 1234ULL},
  {0.00000001, 128, 1024, L2Norm, false, true, 1234ULL},
  {0.00000001, 256, 1024, L2Norm, false, true, 1234ULL},

  {0.00000001, 32, 1024, L1Norm, true, true, 1234ULL},
  {0.00000001, 64, 1024, L1Norm, true, true, 1234ULL},
  {0.00000001, 128, 1024, L1Norm, true, true, 1234ULL},
  {0.00000001, 256, 1024, L1Norm, true, true, 1234ULL},
  {0.00000001, 32, 1024, L2Norm, true, true, 1234ULL},
  {0.00000001, 64, 1024, L2Norm, true, true, 1234ULL},
  {0.00000001, 128, 1024, L2Norm, true, true, 1234ULL},
  {0.00000001, 256, 1024, L2Norm, true, true, 1234ULL}};

typedef ColNormTest<float> ColNormTestF;
TEST_P(ColNormTestF, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef ColNormTest<double> ColNormTestD;
TEST_P(ColNormTestD, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(ColNormTests, ColNormTestF,
                        ::testing::ValuesIn(inputscf));

INSTANTIATE_TEST_CASE_P(ColNormTests, ColNormTestD,
                        ::testing::ValuesIn(inputscd));

}  // end namespace LinAlg
}  // end namespace MLCommon
