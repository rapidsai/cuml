/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>

#include <raft/cudart_utils.h>
#include <raft/linalg/cublas_wrappers.h>
#include <raft/linalg/transpose.h>
#include <raft/cuda_utils.cuh>
#include <raft/linalg/subtract.cuh>
#include <random/make_regression.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Random {

template <typename T>
struct MakeRegressionInputs {
  T tolerance;
  int n_samples, n_features, n_informative, n_targets, effective_rank;
  T bias;
  bool shuffle;
  raft::random::GeneratorType gtype;
  uint64_t seed;
};

template <typename T>
class MakeRegressionTest
  : public ::testing::TestWithParam<MakeRegressionInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MakeRegressionInputs<T>>::GetParam();

    // Noise must be zero to compare the actual and expected values
    T noise = (T)0.0, tail_strength = (T)0.5;

    raft::handle_t handle;
    stream = handle.get_stream();

    raft::allocate(data, params.n_samples * params.n_features);
    raft::allocate(values_ret, params.n_samples * params.n_targets);
    raft::allocate(values_prod, params.n_samples * params.n_targets);
    raft::allocate(values_cm, params.n_samples * params.n_targets);
    raft::allocate(coef, params.n_features * params.n_targets);

    // Create the regression problem
    make_regression(handle, data, values_ret, params.n_samples,
                    params.n_features, params.n_informative, stream, coef,
                    params.n_targets, params.bias, params.effective_rank,
                    tail_strength, noise, params.shuffle, params.seed,
                    params.gtype);

    // Calculate the values from the data and coefficients (column-major)
    T alpha = (T)1.0, beta = (T)0.0;
    CUBLAS_CHECK(raft::linalg::cublasgemm(
      handle.get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, params.n_samples,
      params.n_targets, params.n_features, &alpha, data, params.n_features,
      coef, params.n_targets, &beta, values_cm, params.n_samples, stream));

    // Transpose the values to row-major
    raft::linalg::transpose(handle, values_cm, values_prod, params.n_samples,
                            params.n_targets, stream);

    // Add the bias
    raft::linalg::addScalar(values_prod, values_prod, params.bias,
                            params.n_samples * params.n_targets, stream);

    // Count the number of zeroes in the coefficients
    thrust::device_ptr<T> __coef = thrust::device_pointer_cast(coef);
    zero_count = thrust::count(
      __coef, __coef + params.n_features * params.n_targets, (T)0.0);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(values_ret));
    CUDA_CHECK(hipFree(values_prod));
    CUDA_CHECK(hipFree(values_cm));
  }

 protected:
  MakeRegressionInputs<T> params;
  T *data, *values_ret, *values_prod, *values_cm, *coef;
  int zero_count;
  hipStream_t stream;
};

typedef MakeRegressionTest<float> MakeRegressionTestF;
const std::vector<MakeRegressionInputs<float>> inputsf_t = {
  {0.01f, 256, 32, 16, 1, -1, 0.f, true, raft::random::GenPhilox, 1234ULL},
  {0.01f, 1000, 100, 47, 4, 65, 4.2f, true, raft::random::GenPhilox, 1234ULL},
  {0.01f, 20000, 500, 450, 13, -1, -3.f, false, raft::random::GenPhilox,
   1234ULL}};

TEST_P(MakeRegressionTestF, Result) {
  ASSERT_TRUE(
    match(params.n_targets * (params.n_features - params.n_informative),
          zero_count, raft::Compare<int>()));
  ASSERT_TRUE(
    devArrMatch(values_ret, values_prod, params.n_samples, params.n_targets,
                raft::CompareApprox<float>(params.tolerance), stream));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionTests, MakeRegressionTestF,
                        ::testing::ValuesIn(inputsf_t));

typedef MakeRegressionTest<double> MakeRegressionTestD;
const std::vector<MakeRegressionInputs<double>> inputsd_t = {
  {0.01, 256, 32, 16, 1, -1, 0.0, true, raft::random::GenPhilox, 1234ULL},
  {0.01, 1000, 100, 47, 4, 65, 4.2, true, raft::random::GenPhilox, 1234ULL},
  {0.01, 20000, 500, 450, 13, -1, -3.0, false, raft::random::GenPhilox,
   1234ULL}};

TEST_P(MakeRegressionTestD, Result) {
  ASSERT_TRUE(
    match(params.n_targets * (params.n_features - params.n_informative),
          zero_count, raft::Compare<int>()));
  ASSERT_TRUE(
    devArrMatch(values_ret, values_prod, params.n_samples, params.n_targets,
                raft::CompareApprox<double>(params.tolerance), stream));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionTests, MakeRegressionTestD,
                        ::testing::ValuesIn(inputsd_t));

}  // end namespace Random
}  // end namespace MLCommon
