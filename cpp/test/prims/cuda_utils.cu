#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <cuda_utils.cuh>

namespace MLCommon {

TEST(Utils, Assert) {
  ASSERT_NO_THROW(ASSERT(1 == 1, "Should not assert!"));
  ASSERT_THROW(ASSERT(1 != 1, "Should assert!"), raft::exception);
}

TEST(Utils, CudaCheck) { ASSERT_NO_THROW(CUDA_CHECK(hipFree(nullptr))); }

// we want the functions like 'log2' to work both at compile and runtimes!
static const int log2Of1024 = log2(1024);
static const int log2Of1023 = log2(1023);
TEST(Utils, log2) {
  ASSERT_EQ(10, log2(1024));
  ASSERT_EQ(9, log2(1023));
  ASSERT_EQ(10, log2Of1024);
  ASSERT_EQ(9, log2Of1023);
}

}  // end namespace MLCommon
