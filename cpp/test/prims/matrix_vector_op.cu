/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "matrix_vector_op.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename T, typename IdxType = int>
struct MatVecOpInputs {
  T tolerance;
  IdxType rows, cols;
  bool rowMajor, bcastAlongRows, useTwoVectors;
  unsigned long long int seed;
};

template <typename T, typename IdxType>
::std::ostream &operator<<(::std::ostream &os,
                           const MatVecOpInputs<T, IdxType> &dims) {
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T, typename IdxType>
void matrixVectorOpLaunch(T *out, const T *in, const T *vec1, const T *vec2,
                          IdxType D, IdxType N, bool rowMajor,
                          bool bcastAlongRows, bool useTwoVectors,
                          hipStream_t stream) {
  if (useTwoVectors) {
    matrixVectorOp(
      out, in, vec1, vec2, D, N, rowMajor, bcastAlongRows,
      [] __device__(T a, T b, T c) { return a + b + c; }, stream);
  } else {
    matrixVectorOp(
      out, in, vec1, D, N, rowMajor, bcastAlongRows,
      [] __device__(T a, T b) { return a + b; }, stream);
  }
}

template <typename T, typename IdxType>
class MatVecOpTest
  : public ::testing::TestWithParam<MatVecOpInputs<T, IdxType>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MatVecOpInputs<T, IdxType>>::GetParam();
    Random::Rng r(params.seed);
    IdxType N = params.rows, D = params.cols;
    IdxType len = N * D;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    IdxType vecLen = params.bcastAlongRows ? D : N;
    allocate(vec1, vecLen);
    allocate(vec2, vecLen);
    r.uniform(in, len, (T)-1.0, (T)1.0, stream);
    r.uniform(vec1, vecLen, (T)-1.0, (T)1.0, stream);
    r.uniform(vec2, vecLen, (T)-1.0, (T)1.0, stream);
    if (params.useTwoVectors) {
      naiveMatVec(out_ref, in, vec1, vec2, D, N, params.rowMajor,
                  params.bcastAlongRows, (T)1.0);
    } else {
      naiveMatVec(out_ref, in, vec1, D, N, params.rowMajor,
                  params.bcastAlongRows, (T)1.0);
    }
    matrixVectorOpLaunch(out, in, vec1, vec2, D, N, params.rowMajor,
                         params.bcastAlongRows, params.useTwoVectors, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(vec1));
    CUDA_CHECK(hipFree(vec2));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(in));
  }

 protected:
  MatVecOpInputs<T, IdxType> params;
  T *in, *out, *out_ref, *vec1, *vec2;
};

const std::vector<MatVecOpInputs<float, int>> inputsf_i32 = {
  {0.00001f, 1024, 32, true, true, false, 1234ULL},
  {0.00001f, 1024, 64, true, true, false, 1234ULL},
  {0.00001f, 1024, 32, true, false, false, 1234ULL},
  {0.00001f, 1024, 64, true, false, false, 1234ULL},
  {0.00001f, 1024, 32, false, true, false, 1234ULL},
  {0.00001f, 1024, 64, false, true, false, 1234ULL},
  {0.00001f, 1024, 32, false, false, false, 1234ULL},
  {0.00001f, 1024, 64, false, false, false, 1234ULL},

  {0.00001f, 1024, 32, true, true, true, 1234ULL},
  {0.00001f, 1024, 64, true, true, true, 1234ULL},
  {0.00001f, 1024, 32, true, false, true, 1234ULL},
  {0.00001f, 1024, 64, true, false, true, 1234ULL},
  {0.00001f, 1024, 32, false, true, true, 1234ULL},
  {0.00001f, 1024, 64, false, true, true, 1234ULL},
  {0.00001f, 1024, 32, false, false, true, 1234ULL},
  {0.00001f, 1024, 64, false, false, true, 1234ULL}};
typedef MatVecOpTest<float, int> MatVecOpTestF_i32;
TEST_P(MatVecOpTestF_i32, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestF_i32,
                        ::testing::ValuesIn(inputsf_i32));

const std::vector<MatVecOpInputs<float, size_t>> inputsf_i64 = {
  {0.00001f, 2500, 250, false, false, false, 1234ULL},
  {0.00001f, 2500, 250, false, false, true, 1234ULL}};
typedef MatVecOpTest<float, size_t> MatVecOpTestF_i64;
TEST_P(MatVecOpTestF_i64, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestF_i64,
                        ::testing::ValuesIn(inputsf_i64));

const std::vector<MatVecOpInputs<double, int>> inputsd_i32 = {
  {0.0000001, 1024, 32, true, true, false, 1234ULL},
  {0.0000001, 1024, 64, true, true, false, 1234ULL},
  {0.0000001, 1024, 32, true, false, false, 1234ULL},
  {0.0000001, 1024, 64, true, false, false, 1234ULL},
  {0.0000001, 1024, 32, false, true, false, 1234ULL},
  {0.0000001, 1024, 64, false, true, false, 1234ULL},
  {0.0000001, 1024, 32, false, false, false, 1234ULL},
  {0.0000001, 1024, 64, false, false, false, 1234ULL},

  {0.0000001, 1024, 32, true, true, true, 1234ULL},
  {0.0000001, 1024, 64, true, true, true, 1234ULL},
  {0.0000001, 1024, 32, true, false, true, 1234ULL},
  {0.0000001, 1024, 64, true, false, true, 1234ULL},
  {0.0000001, 1024, 32, false, true, true, 1234ULL},
  {0.0000001, 1024, 64, false, true, true, 1234ULL},
  {0.0000001, 1024, 32, false, false, true, 1234ULL},
  {0.0000001, 1024, 64, false, false, true, 1234ULL}};
typedef MatVecOpTest<double, int> MatVecOpTestD_i32;
TEST_P(MatVecOpTestD_i32, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestD_i32,
                        ::testing::ValuesIn(inputsd_i32));

const std::vector<MatVecOpInputs<double, size_t>> inputsd_i64 = {
  {0.0000001, 2500, 250, false, false, false, 1234ULL},
  {0.0000001, 2500, 250, false, false, true, 1234ULL}};
typedef MatVecOpTest<double, size_t> MatVecOpTestD_i64;
TEST_P(MatVecOpTestD_i64, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.rows * params.cols,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(MatVecOpTests, MatVecOpTestD_i64,
                        ::testing::ValuesIn(inputsd_i64));

}  // end namespace LinAlg
}  // end namespace MLCommon
