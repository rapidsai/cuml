/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "linalg/unary_op.h"
#include "random/rng.h"
#include "test_utils.h"
#include "unary_op.h"


namespace MLCommon {
namespace LinAlg {

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T, typename IdxType = int>
void unaryOpLaunch(T *out, const T *in, T scalar, IdxType len, hipStream_t stream) {
  unaryOp(out, in, len,
          [scalar] __device__(T in) { return in * scalar; },
          stream);
}

template <typename T, typename IdxType>
class UnaryOpTest : public ::testing::TestWithParam<UnaryOpInputs<T, IdxType>> {
protected:
  void SetUp() override {
    params = ::testing::TestWithParam<UnaryOpInputs<T, IdxType>>::GetParam();
    Random::Rng r(params.seed);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    auto len = params.len;
    auto scalar = params.scalar;

    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in, len, T(-1.0), T(1.0), stream);
    naiveScale(out_ref, in, scalar, len, stream);
    unaryOpLaunch(out, in, scalar, len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

protected:
  UnaryOpInputs<T, IdxType> params;
  T *in, *out_ref, *out;
};

const std::vector<UnaryOpInputs<float, int>> inputsf_i32 = {
  {0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef UnaryOpTest<float, int> UnaryOpTestF_i32;
TEST_P(UnaryOpTestF_i32, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(UnaryOpTests, UnaryOpTestF_i32,
                        ::testing::ValuesIn(inputsf_i32));

const std::vector<UnaryOpInputs<float, size_t>> inputsf_i64 = {
  {0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef UnaryOpTest<float, size_t> UnaryOpTestF_i64;
TEST_P(UnaryOpTestF_i64, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(UnaryOpTests, UnaryOpTestF_i64,
                        ::testing::ValuesIn(inputsf_i64));

const std::vector<UnaryOpInputs<double, int>> inputsd_i32 = {
  {0.00000001, 1024 * 1024, 2.0, 1234ULL}};
typedef UnaryOpTest<double, int> UnaryOpTestD_i32;
TEST_P(UnaryOpTestD_i32, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(UnaryOpTests, UnaryOpTestD_i32,
                        ::testing::ValuesIn(inputsd_i32));

const std::vector<UnaryOpInputs<double, size_t>> inputsd_i64 = {
  {0.00000001, 1024 * 1024, 2.0, 1234ULL}};
typedef UnaryOpTest<double, size_t> UnaryOpTestD_i64;
TEST_P(UnaryOpTestD_i64, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(UnaryOpTests, UnaryOpTestD_i64,
                        ::testing::ValuesIn(inputsd_i64));

} // end namespace LinAlg
} // end namespace MLCommon
