/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/unary_op.h"
#include "random/rng.h"
#include "test_utils.h"
#include "unary_op.h"

namespace MLCommon {
namespace LinAlg {

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename InType, typename IdxType = int, typename OutType = InType>
void unaryOpLaunch(OutType *out, const InType *in, InType scalar, IdxType len,
                   hipStream_t stream) {
  if (in == nullptr) {
    auto op = [scalar] __device__(OutType * ptr, IdxType idx) {
      *ptr = static_cast<OutType>(scalar * idx);
    };
    writeOnlyUnaryOp<OutType, decltype(op), IdxType>(out, len, op, stream);
  } else {
    auto op = [scalar] __device__(InType in) {
      return static_cast<OutType>(in * scalar);
    };
    unaryOp<InType, decltype(op), IdxType, OutType>(out, in, len, op, stream);
  }
}

template <typename InType, typename IdxType, typename OutType = InType>
class UnaryOpTest
  : public ::testing::TestWithParam<UnaryOpInputs<InType, IdxType, OutType>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<
      UnaryOpInputs<InType, IdxType, OutType>>::GetParam();
    Random::Rng r(params.seed);
    CUDA_CHECK(hipStreamCreate(&stream));
    auto len = params.len;
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in, len, InType(-1.0), InType(1.0), stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

  virtual void DoTest() {
    auto len = params.len;
    auto scalar = params.scalar;
    naiveScale(out_ref, in, scalar, len, stream);
    unaryOpLaunch(out, in, scalar, len, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                            CompareApprox<OutType>(params.tolerance)));
  }

  UnaryOpInputs<InType, IdxType, OutType> params;
  InType *in;
  OutType *out_ref, *out;
  hipStream_t stream;
};

template <typename OutType, typename IdxType>
class WriteOnlyUnaryOpTest : public UnaryOpTest<OutType, IdxType, OutType> {
 protected:
  void DoTest() override {
    auto len = this->params.len;
    auto scalar = this->params.scalar;
    naiveScale(this->out_ref, (OutType *)nullptr, scalar, len, this->stream);
    unaryOpLaunch(this->out, (OutType *)nullptr, scalar, len, this->stream);
    CUDA_CHECK(hipStreamSynchronize(this->stream));
    ASSERT_TRUE(devArrMatch(this->out_ref, this->out, this->params.len,
                            CompareApprox<OutType>(this->params.tolerance)));
  }
};

#define UNARY_OP_TEST(Name, inputs)  \
  TEST_P(Name, Result) { DoTest(); } \
  INSTANTIATE_TEST_CASE_P(UnaryOpTests, Name, ::testing::ValuesIn(inputs))

const std::vector<UnaryOpInputs<float, int>> inputsf_i32 = {
  {0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef UnaryOpTest<float, int> UnaryOpTestF_i32;
UNARY_OP_TEST(UnaryOpTestF_i32, inputsf_i32);
typedef WriteOnlyUnaryOpTest<float, int> WriteOnlyUnaryOpTestF_i32;
UNARY_OP_TEST(WriteOnlyUnaryOpTestF_i32, inputsf_i32);

const std::vector<UnaryOpInputs<float, size_t>> inputsf_i64 = {
  {0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef UnaryOpTest<float, size_t> UnaryOpTestF_i64;
UNARY_OP_TEST(UnaryOpTestF_i64, inputsf_i64);
typedef WriteOnlyUnaryOpTest<float, size_t> WriteOnlyUnaryOpTestF_i64;
UNARY_OP_TEST(WriteOnlyUnaryOpTestF_i64, inputsf_i64);

const std::vector<UnaryOpInputs<float, int, double>> inputsf_i32_d = {
  {0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef UnaryOpTest<float, int, double> UnaryOpTestF_i32_D;
UNARY_OP_TEST(UnaryOpTestF_i32_D, inputsf_i32_d);

const std::vector<UnaryOpInputs<double, int>> inputsd_i32 = {
  {0.00000001, 1024 * 1024, 2.0, 1234ULL}};
typedef UnaryOpTest<double, int> UnaryOpTestD_i32;
UNARY_OP_TEST(UnaryOpTestD_i32, inputsd_i32);
typedef WriteOnlyUnaryOpTest<double, int> WriteOnlyUnaryOpTestD_i32;
UNARY_OP_TEST(WriteOnlyUnaryOpTestD_i32, inputsd_i32);

const std::vector<UnaryOpInputs<double, size_t>> inputsd_i64 = {
  {0.00000001, 1024 * 1024, 2.0, 1234ULL}};
typedef UnaryOpTest<double, size_t> UnaryOpTestD_i64;
UNARY_OP_TEST(UnaryOpTestD_i64, inputsd_i64);
typedef WriteOnlyUnaryOpTest<double, size_t> WriteOnlyUnaryOpTestD_i64;
UNARY_OP_TEST(WriteOnlyUnaryOpTestD_i64, inputsd_i64);

}  // end namespace LinAlg
}  // end namespace MLCommon
