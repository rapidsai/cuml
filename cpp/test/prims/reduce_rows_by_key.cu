#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <iostream>
#include "linalg/reduce_rows_by_key.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename Type>
__global__ void naiveReduceRowsByKeyKernel(Type *d_A, int lda, uint32_t *d_keys,
                                           char *d_char_keys, int nrows,
                                           int ncols, int nkeys, Type *d_sums) {
  int c = threadIdx.x + blockIdx.x * blockDim.x;
  if (c >= ncols) return;
  int this_key = threadIdx.y + blockIdx.y * blockDim.y;

  Type sum = 0.0;
  for (int r = 0; r < nrows; r++) {
    if (this_key != d_keys[r]) continue;
    sum += d_A[lda * r + c];
  }
  d_sums[this_key * ncols + c] = sum;
}
template <typename Type>
void naiveReduceRowsByKey(Type *d_A, int lda, uint32_t *d_keys,
                          char *d_char_keys, int nrows, int ncols, int nkeys,
                          Type *d_sums, hipStream_t stream) {
  hipMemset(d_sums, 0, sizeof(Type) * nkeys * ncols);

  naiveReduceRowsByKeyKernel<<<dim3((ncols + 31) / 32, nkeys), dim3(32, 1), 0,
                               stream>>>(d_A, lda, d_keys, d_char_keys, nrows,
                                         ncols, nkeys, d_sums);
}

template <typename T>
struct ReduceRowsInputs {
  T tolerance;
  int nobs;
  uint32_t cols;
  uint32_t nkeys;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os,
                           const ReduceRowsInputs<T> &dims) {
  return os;
}

template <typename T>
class ReduceRowTest : public ::testing::TestWithParam<ReduceRowsInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<ReduceRowsInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    Random::Rng r_int(params.seed);
    CUDA_CHECK(hipStreamCreate(&stream));

    int nobs = params.nobs;
    uint32_t cols = params.cols;
    uint32_t nkeys = params.nkeys;
    allocate(in1, nobs * cols);
    allocate(in2, nobs);
    allocate(chars2, nobs);
    allocate(out_ref, nkeys * cols);
    allocate(out, nkeys * cols);
    r.uniform(in1, nobs * cols, T(0.0), T(2.0 / nobs), stream);
    r_int.uniformInt(in2, nobs, (uint32_t)0, nkeys, stream);
    naiveReduceRowsByKey(in1, cols, in2, chars2, nobs, cols, nkeys, out_ref,
                         stream);
    reduce_rows_by_key(in1, cols, in2, chars2, nobs, cols, nkeys, out, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    CUDA_CHECK(hipFree(chars2));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  hipStream_t stream;
  ReduceRowsInputs<T> params;
  T *in1, *out_ref, *out, *out_2;
  uint32_t *in2;
  char *chars2;
  int device_count = 0;
};

// ReduceRowTestF
// 128 Obs, 32 cols, 6 clusters
const std::vector<ReduceRowsInputs<float>> inputsf2 = {
  {0.000001f, 128, 32, 6, 1234ULL}};
typedef ReduceRowTest<float> ReduceRowTestF;
TEST_P(ReduceRowTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.cols * params.nkeys,
                          CompareApprox<float>(params.tolerance)));
  //if (device_count > 1) {
  // 	ASSERT_TRUE(devArrMatch(out_ref, out_2, params.len,
  //	                    CompareApprox<double>(params.tolerance)));
  //}
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestF,
                        ::testing::ValuesIn(inputsf2));

// ReduceRowTestD
// 128 Obs, 32 cols, 6 clusters, double precision
const std::vector<ReduceRowsInputs<double>> inputsd2 = {
  {0.00000001, 128, 32, 6, 1234ULL}};
typedef ReduceRowTest<double> ReduceRowTestD;
TEST_P(ReduceRowTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.cols * params.nkeys,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestD,
                        ::testing::ValuesIn(inputsd2));

// ReduceRowTestSmallnKey
// 128 Obs, 32 cols, 3 clusters
const std::vector<ReduceRowsInputs<float>> inputsf_small_nkey = {
  {0.000001f, 128, 32, 3, 1234ULL}};
typedef ReduceRowTest<float> ReduceRowTestSmallnKey;
TEST_P(ReduceRowTestSmallnKey, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.cols * params.nkeys,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestSmallnKey,
                        ::testing::ValuesIn(inputsf_small_nkey));

// ReduceRowTestBigSpace
// 512 Obs, 1024 cols, 32 clusters, double precision
const std::vector<ReduceRowsInputs<double>> inputsd_big_space = {
  {0.00000001, 512, 1024, 40, 1234ULL}};
typedef ReduceRowTest<double> ReduceRowTestBigSpace;
TEST_P(ReduceRowTestBigSpace, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.cols * params.nkeys,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestBigSpace,
                        ::testing::ValuesIn(inputsd_big_space));

// ReduceRowTestManyObs
// 100000 Obs, 37 cols, 32 clusters
const std::vector<ReduceRowsInputs<float>> inputsf_many_obs = {
  {0.00001f, 100000, 37, 32, 1234ULL}};
typedef ReduceRowTest<float> ReduceRowTestManyObs;
TEST_P(ReduceRowTestManyObs, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.cols * params.nkeys,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestManyObs,
                        ::testing::ValuesIn(inputsf_many_obs));

// ReduceRowTestManyClusters
// 100000 Obs, 37 cols, 2048 clusters
const std::vector<ReduceRowsInputs<float>> inputsf_many_cluster = {
  {0.00001f, 100000, 37, 2048, 1234ULL}};
typedef ReduceRowTest<float> ReduceRowTestManyClusters;
TEST_P(ReduceRowTestManyClusters, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.cols * params.nkeys,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestManyClusters,
                        ::testing::ValuesIn(inputsf_many_cluster));

}  // end namespace LinAlg
}  // end namespace MLCommon
