#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <cuda_utils.h>
#include <distance/fused_l2_nn.h>
#include <gtest/gtest.h>
#include <linalg/norm.h>
#include <random/rng.h>
#include "test_utils.h"

namespace MLCommon {
namespace Distance {

template <typename DataT, bool Sqrt, typename ReduceOpT, int NWARPS>
__global__ void naiveKernel(hipcub::KeyValuePair<int, DataT> *min, DataT *x,
                            DataT *y, int m, int n, int k, int *workspace,
                            DataT maxVal) {
  int midx = threadIdx.y + blockIdx.y * blockDim.y;
  int nidx = threadIdx.x + blockIdx.x * blockDim.x;
  DataT acc = DataT(0);
  for (int i = 0; i < k; ++i) {
    int xidx = i + midx * k;
    int yidx = i + nidx * k;
    auto diff = midx >= m || nidx >= n ? DataT(0) : x[xidx] - y[yidx];
    acc += diff * diff;
  }
  if (Sqrt) {
    acc = mySqrt(acc);
  }
  ReduceOpT redOp;
  typedef hipcub::WarpReduce<hipcub::KeyValuePair<int, DataT>> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp[NWARPS];
  int warpId = threadIdx.x / WarpSize;
  hipcub::KeyValuePair<int, DataT> tmp;
  tmp.key = nidx;
  tmp.value = midx >= m || nidx >= n ? maxVal : acc;
  tmp = WarpReduce(temp[warpId]).Reduce(tmp, KVPMinReduce<int, DataT>());
  if (threadIdx.x % WarpSize == 0 && midx < m) {
    while (atomicCAS(workspace + midx, 0, 1) == 1)
      ;
    __threadfence();
    redOp(min + midx, tmp);
    __threadfence();
    atomicCAS(workspace + midx, 1, 0);
  }
}

template <typename DataT, bool Sqrt>
void naive(hipcub::KeyValuePair<int, DataT> *min, DataT *x, DataT *y, int m, int n,
           int k, int *workspace, hipStream_t stream) {
  static const dim3 TPB(32, 16, 1);
  dim3 nblks(ceildiv(n, (int)TPB.x), ceildiv(m, (int)TPB.y), 1);
  CUDA_CHECK(hipMemsetAsync(workspace, 0, sizeof(int) * m, stream));
  auto blks = ceildiv(m, 256);
  MinAndDistanceReduceOp<int, DataT> op;
  initKernel<DataT, hipcub::KeyValuePair<int, DataT>, int>
    <<<blks, 256, 0, stream>>>(min, m, std::numeric_limits<DataT>::max(), op);
  CUDA_CHECK(hipGetLastError());
  naiveKernel<DataT, Sqrt, MinAndDistanceReduceOp<int, DataT>, 16>
    <<<nblks, TPB, 0, stream>>>(min, x, y, m, n, k, workspace,
                                std::numeric_limits<DataT>::max());
  CUDA_CHECK(hipGetLastError());
}

template <typename DataT>
struct Inputs {
  DataT tolerance;
  int m, n, k;
  unsigned long long int seed;
};

template <typename DataT, bool Sqrt>
class FusedL2NNTest : public ::testing::TestWithParam<Inputs<DataT>> {
 public:
  void SetUp() override {
    params = ::testing::TestWithParam<Inputs<DataT>>::GetParam();
    Random::Rng r(params.seed);
    int m = params.m;
    int n = params.n;
    int k = params.k;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(x, m * k);
    allocate(y, n * k);
    allocate(xn, m);
    allocate(yn, n);
    allocate(workspace, sizeof(int) * m);
    allocate(min, m);
    allocate(min_ref, m);
    r.uniform(x, m * k, DataT(-1.0), DataT(1.0), stream);
    r.uniform(y, n * k, DataT(-1.0), DataT(1.0), stream);
    generateGoldenResult();
    LinAlg::rowNorm(xn, x, k, m, LinAlg::L2Norm, true, stream);
    LinAlg::rowNorm(yn, y, k, n, LinAlg::L2Norm, true, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(x));
    CUDA_CHECK(hipFree(y));
    CUDA_CHECK(hipFree(xn));
    CUDA_CHECK(hipFree(yn));
    CUDA_CHECK(hipFree(workspace));
    CUDA_CHECK(hipFree(min_ref));
    CUDA_CHECK(hipFree(min));
  }

 protected:
  Inputs<DataT> params;
  DataT *x, *y, *xn, *yn;
  char *workspace;
  hipcub::KeyValuePair<int, DataT> *min, *min_ref;
  hipStream_t stream;

  virtual void generateGoldenResult() {
    int m = params.m;
    int n = params.n;
    int k = params.k;
    naive<DataT, Sqrt>(min_ref, x, y, m, n, k, (int *)workspace, stream);
  }

  void runTest(hipcub::KeyValuePair<int, DataT> *out) {
    int m = params.m;
    int n = params.n;
    int k = params.k;
    MinAndDistanceReduceOp<int, DataT> redOp;
    fusedL2NN<DataT, hipcub::KeyValuePair<int, DataT>, int>(
      out, x, y, xn, yn, m, n, k, (void *)workspace, redOp, Sqrt, true, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
};

template <typename T>
struct CompareApproxAbsKVP {
  typedef typename hipcub::KeyValuePair<int, T> KVP;
  CompareApproxAbsKVP(T eps_) : eps(eps_) {}
  bool operator()(const KVP &a, const KVP &b) const {
    if (a.key != b.key) return false;
    T diff = abs(abs(a.value) - abs(b.value));
    T m = std::max(abs(a.value), abs(b.value));
    T ratio = m >= eps ? diff / m : diff;
    return (ratio <= eps);
  }

 private:
  T eps;
};

template <typename T>
struct CompareExactKVP {
  typedef typename hipcub::KeyValuePair<int, T> KVP;
  bool operator()(const KVP &a, const KVP &b) const {
    if (a.key != b.key) return false;
    if (a.value != b.value) return false;
    return true;
  }
};

template <typename K, typename V, typename L>
::testing::AssertionResult devArrMatch(const hipcub::KeyValuePair<K, V> *expected,
                                       const hipcub::KeyValuePair<K, V> *actual,
                                       size_t size, L eq_compare,
                                       hipStream_t stream = 0) {
  typedef typename hipcub::KeyValuePair<K, V> KVP;
  std::shared_ptr<KVP> exp_h(new KVP[size]);
  std::shared_ptr<KVP> act_h(new KVP[size]);
  updateHost<KVP>(exp_h.get(), expected, size, stream);
  updateHost<KVP>(act_h.get(), actual, size, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  for (size_t i(0); i < size; ++i) {
    auto exp = exp_h.get()[i];
    auto act = act_h.get()[i];
    if (!eq_compare(exp, act)) {
      return ::testing::AssertionFailure()
             << "actual=" << act.key << "," << act.value
             << " != expected=" << exp.key << "," << exp.value << " @" << i;
    }
  }
  return ::testing::AssertionSuccess();
}

const std::vector<Inputs<float>> inputsf = {
  {0.001f, 32, 32, 32, 1234ULL},   {0.001f, 32, 64, 32, 1234ULL},
  {0.001f, 64, 32, 32, 1234ULL},   {0.001f, 64, 64, 32, 1234ULL},
  {0.001f, 128, 32, 32, 1234ULL},  {0.001f, 128, 64, 32, 1234ULL},
  {0.001f, 128, 128, 64, 1234ULL}, {0.001f, 64, 128, 128, 1234ULL},

  {0.001f, 32, 32, 34, 1234ULL},   {0.001f, 32, 64, 34, 1234ULL},
  {0.001f, 64, 32, 34, 1234ULL},   {0.001f, 64, 64, 34, 1234ULL},
  {0.001f, 128, 32, 34, 1234ULL},  {0.001f, 128, 64, 34, 1234ULL},
  {0.001f, 128, 128, 66, 1234ULL}, {0.001f, 64, 128, 130, 1234ULL},

  {0.001f, 32, 32, 33, 1234ULL},   {0.001f, 32, 64, 33, 1234ULL},
  {0.001f, 64, 32, 33, 1234ULL},   {0.001f, 64, 64, 33, 1234ULL},
  {0.001f, 128, 32, 33, 1234ULL},  {0.001f, 128, 64, 33, 1234ULL},
  {0.001f, 128, 128, 65, 1234ULL}, {0.001f, 64, 128, 129, 1234ULL},

  {0.006f, 1805, 134, 2, 1234ULL},
};
typedef FusedL2NNTest<float, false> FusedL2NNTestF_Sq;
TEST_P(FusedL2NNTestF_Sq, Result) {
  runTest(min);
  ASSERT_TRUE(devArrMatch(min_ref, min, params.m,
                          CompareApproxAbsKVP<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestF_Sq,
                        ::testing::ValuesIn(inputsf));
typedef FusedL2NNTest<float, true> FusedL2NNTestF_Sqrt;
TEST_P(FusedL2NNTestF_Sqrt, Result) {
  runTest(min);
  ASSERT_TRUE(devArrMatch(min_ref, min, params.m,
                          CompareApproxAbsKVP<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestF_Sqrt,
                        ::testing::ValuesIn(inputsf));

const std::vector<Inputs<double>> inputsd = {
  {0.00001, 32, 32, 32, 1234ULL},   {0.00001, 32, 64, 32, 1234ULL},
  {0.00001, 64, 32, 32, 1234ULL},   {0.00001, 64, 64, 32, 1234ULL},
  {0.00001, 128, 32, 32, 1234ULL},  {0.00001, 128, 64, 32, 1234ULL},
  {0.00001, 128, 128, 64, 1234ULL}, {0.00001, 64, 128, 128, 1234ULL},

  {0.00001, 32, 32, 34, 1234ULL},   {0.00001, 32, 64, 34, 1234ULL},
  {0.00001, 64, 32, 34, 1234ULL},   {0.00001, 64, 64, 34, 1234ULL},
  {0.00001, 128, 32, 34, 1234ULL},  {0.00001, 128, 64, 34, 1234ULL},
  {0.00001, 128, 128, 66, 1234ULL}, {0.00001, 64, 128, 130, 1234ULL},

  {0.00001, 32, 32, 33, 1234ULL},   {0.00001, 32, 64, 33, 1234ULL},
  {0.00001, 64, 32, 33, 1234ULL},   {0.00001, 64, 64, 33, 1234ULL},
  {0.00001, 128, 32, 33, 1234ULL},  {0.00001, 128, 64, 33, 1234ULL},
  {0.00001, 128, 128, 65, 1234ULL}, {0.00001, 64, 128, 129, 1234ULL},

  {0.00001, 1805, 134, 2, 1234ULL},
};
typedef FusedL2NNTest<double, false> FusedL2NNTestD_Sq;
TEST_P(FusedL2NNTestD_Sq, Result) {
  runTest(min);
  ASSERT_TRUE(devArrMatch(min_ref, min, params.m,
                          CompareApproxAbsKVP<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestD_Sq,
                        ::testing::ValuesIn(inputsd));
typedef FusedL2NNTest<double, true> FusedL2NNTestD_Sqrt;
TEST_P(FusedL2NNTestD_Sqrt, Result) {
  runTest(min);
  ASSERT_TRUE(devArrMatch(min_ref, min, params.m,
                          CompareApproxAbsKVP<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestD_Sqrt,
                        ::testing::ValuesIn(inputsd));

/// This is to test output determinism of the prim
template <typename DataT, bool Sqrt>
class FusedL2NNDetTest : public FusedL2NNTest<DataT, Sqrt> {
  void SetUp() override {
    FusedL2NNTest<DataT, Sqrt>::SetUp();
    int m = this->params.m;
    allocate(min1, m);
  }

  void TearDown() override {
    FusedL2NNTest<DataT, Sqrt>::TearDown();
    CUDA_CHECK(hipFree(min1));
  }

 protected:
  hipcub::KeyValuePair<int, DataT> *min1;

  static const int NumRepeats = 100;

  void generateGoldenResult() override {}
};

typedef FusedL2NNDetTest<float, false> FusedL2NNDetTestF_Sq;
TEST_P(FusedL2NNDetTestF_Sq, Result) {
  runTest(min);  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1);
    ASSERT_TRUE(devArrMatch(min, min1, params.m, CompareExactKVP<float>()));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestF_Sq,
                        ::testing::ValuesIn(inputsf));
typedef FusedL2NNDetTest<float, true> FusedL2NNDetTestF_Sqrt;
TEST_P(FusedL2NNDetTestF_Sqrt, Result) {
  runTest(min);  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1);
    ASSERT_TRUE(devArrMatch(min, min1, params.m, CompareExactKVP<float>()));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestF_Sqrt,
                        ::testing::ValuesIn(inputsf));

typedef FusedL2NNDetTest<double, false> FusedL2NNDetTestD_Sq;
TEST_P(FusedL2NNDetTestD_Sq, Result) {
  runTest(min);  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1);
    ASSERT_TRUE(devArrMatch(min, min1, params.m, CompareExactKVP<double>()));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestD_Sq,
                        ::testing::ValuesIn(inputsd));
typedef FusedL2NNDetTest<double, true> FusedL2NNDetTestD_Sqrt;
TEST_P(FusedL2NNDetTestD_Sqrt, Result) {
  runTest(min);  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1);
    ASSERT_TRUE(devArrMatch(min, min1, params.m, CompareExactKVP<double>()));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestD_Sqrt,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Distance
}  // end namespace MLCommon
