/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "test_utils.h"
#include <gtest/gtest.h>
#include <iostream>
#include <raft/cuda_utils.cuh>
#include <raft/cudart_utils.h>
#include <raft/label/classlabels.hpp>
#include <raft/random/make_blobs.hpp>
#include <raft/spatial/knn/knn.hpp>
#include <rmm/device_uvector.hpp>
#include <selection/knn.cuh>
#include <vector>

namespace MLCommon {
namespace Selection {

struct KNNClassifyInputs {
  int rows;
  int cols;
  int n_labels;
  float cluster_std;
  int k;
};

class KNNClassifyTest : public ::testing::TestWithParam<KNNClassifyInputs> {
 public:
  KNNClassifyTest()
    : params(::testing::TestWithParam<KNNClassifyInputs>::GetParam()),
      stream(handle.get_stream()),
      train_samples(params.rows * params.cols, stream),
      train_labels(params.rows, stream),
      pred_labels(params.rows, stream),
      knn_indices(params.rows * params.k, stream),
      knn_dists(params.rows * params.k, stream)
  {
    basicTest();
  }

 protected:
  void basicTest()
  {
    raft::random::make_blobs<float, int>(train_samples.data(),
                                         train_labels.data(),
                                         params.rows,
                                         params.cols,
                                         params.n_labels,
                                         stream,
                                         true,
                                         nullptr,
                                         nullptr,
                                         params.cluster_std);

    rmm::device_uvector<int> unique_labels(0, stream);
    auto n_classes =
      raft::label::getUniquelabels(unique_labels, train_labels.data(), params.rows, stream);

    std::vector<float*> ptrs(1);
    std::vector<int> sizes(1);
    ptrs[0]  = train_samples.data();
    sizes[0] = params.rows;

    raft::spatial::knn::brute_force_knn(handle,
                                        ptrs,
                                        sizes,
                                        params.cols,
                                        train_samples.data(),
                                        params.rows,
                                        knn_indices.data(),
                                        knn_dists.data(),
                                        params.k);

    std::vector<int*> y;
    y.push_back(train_labels.data());

    std::vector<int*> uniq_labels;
    uniq_labels.push_back(unique_labels.data());

    std::vector<int> n_unique;
    n_unique.push_back(n_classes);

    knn_classify(handle,
                 pred_labels.data(),
                 knn_indices.data(),
                 y,
                 params.rows,
                 params.rows,
                 params.k,
                 uniq_labels,
                 n_unique);

    handle.sync_stream(stream);
  }

 protected:
  KNNClassifyInputs params;
  raft::handle_t handle;
  hipStream_t stream;

  rmm::device_uvector<float> train_samples;
  rmm::device_uvector<int> train_labels;

  rmm::device_uvector<int> pred_labels;

  rmm::device_uvector<int64_t> knn_indices;
  rmm::device_uvector<float> knn_dists;
};

typedef KNNClassifyTest KNNClassifyTestF;
TEST_P(KNNClassifyTestF, Fit)
{
  ASSERT_TRUE(
    devArrMatch(train_labels.data(), pred_labels.data(), params.rows, raft::Compare<int>()));
}

const std::vector<KNNClassifyInputs> inputsf = {{100, 10, 2, 0.01f, 2},
                                                {1000, 10, 5, 0.01f, 2},
                                                {10000, 10, 5, 0.01f, 2},
                                                {100, 10, 2, 0.01f, 10},
                                                {1000, 10, 5, 0.01f, 10},
                                                {10000, 10, 5, 0.01f, 10},
                                                {100, 10, 2, 0.01f, 50},
                                                {1000, 10, 5, 0.01f, 50},
                                                {10000, 10, 5, 0.01f, 50}};

INSTANTIATE_TEST_CASE_P(KNNClassifyTest, KNNClassifyTestF, ::testing::ValuesIn(inputsf));

};  // end namespace Selection
};  // namespace MLCommon
