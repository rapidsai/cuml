/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <iostream>
#include <label/classlabels.cuh>
#include <raft/cuda_utils.cuh>
#include <random/make_blobs.cuh>
#include <selection/knn.cuh>
#include <vector>
#include "test_utils.h"

namespace MLCommon {
namespace Selection {

struct KNNClassifyInputs {
  int rows;
  int cols;
  int n_labels;
  float cluster_std;
  int k;
};

class KNNClassifyTest : public ::testing::TestWithParam<KNNClassifyInputs> {
 protected:
  void basicTest() {
    std::shared_ptr<MLCommon::deviceAllocator> alloc(
      new raft::mr::device::default_allocator);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    params = ::testing::TestWithParam<KNNClassifyInputs>::GetParam();

    raft::allocate(train_samples, params.rows * params.cols);
    raft::allocate(train_labels, params.rows);

    raft::allocate(pred_labels, params.rows);
    raft::allocate(unique_labels, params.n_labels, true);

    raft::allocate(knn_indices, params.rows * params.k);
    raft::allocate(knn_dists, params.rows * params.k);

    MLCommon::Random::make_blobs<float, int>(
      train_samples, train_labels, params.rows, params.cols, params.n_labels,
      alloc, stream, true, nullptr, nullptr, params.cluster_std);

    int n_classes;
    MLCommon::Label::getUniqueLabels(train_labels, params.rows, &unique_labels,
                                     &n_classes, stream, alloc);

    std::vector<float *> ptrs(1);
    std::vector<int> sizes(1);
    ptrs[0] = train_samples;
    sizes[0] = params.rows;

    brute_force_knn(ptrs, sizes, params.cols, train_samples, params.rows,
                    knn_indices, knn_dists, params.k, alloc, stream);

    std::vector<int *> y;
    y.push_back(train_labels);

    std::vector<int *> uniq_labels;
    uniq_labels.push_back(unique_labels);

    std::vector<int> n_unique;
    n_unique.push_back(n_classes);

    knn_classify(pred_labels, knn_indices, y, params.rows, params.rows,
                 params.k, uniq_labels, n_unique, alloc, stream);

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    CUDA_CHECK(hipFree(train_samples));
    CUDA_CHECK(hipFree(train_labels));

    CUDA_CHECK(hipFree(pred_labels));

    CUDA_CHECK(hipFree(knn_indices));
    CUDA_CHECK(hipFree(knn_dists));

    CUDA_CHECK(hipFree(unique_labels));
  }

 protected:
  KNNClassifyInputs params;

  float *train_samples;
  int *train_labels;

  int *pred_labels;

  int64_t *knn_indices;
  float *knn_dists;

  int *unique_labels;
};

typedef KNNClassifyTest KNNClassifyTestF;
TEST_P(KNNClassifyTestF, Fit) {
  ASSERT_TRUE(
    devArrMatch(train_labels, pred_labels, params.rows, raft::Compare<int>()));
}

const std::vector<KNNClassifyInputs> inputsf = {
  {100, 10, 2, 0.01f, 2},  {1000, 10, 5, 0.01f, 2},  {10000, 10, 5, 0.01f, 2},
  {100, 10, 2, 0.01f, 10}, {1000, 10, 5, 0.01f, 10}, {10000, 10, 5, 0.01f, 10},
  {100, 10, 2, 0.01f, 50}, {1000, 10, 5, 0.01f, 50}, {10000, 10, 5, 0.01f, 50}};

INSTANTIATE_TEST_CASE_P(KNNClassifyTest, KNNClassifyTestF,
                        ::testing::ValuesIn(inputsf));

};  // end namespace Selection
};  // namespace MLCommon
