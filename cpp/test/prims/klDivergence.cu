#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <algorithm>
#include <cuml/common/cuml_allocator.hpp>
#include <iostream>
#include <random>
#include "metrics/klDivergence.h"
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

//parameter structure definition
struct klDivergenceParam {
  int nElements;
  double tolerance;
};

//test fixture class
template <typename DataT>
class klDivergenceTest : public ::testing::TestWithParam<klDivergenceParam> {
 protected:
  //the constructor
  void SetUp() override {
    //getting the parameters
    params = ::testing::TestWithParam<klDivergenceParam>::GetParam();

    nElements = params.nElements;

    //generating random value test input
    std::vector<DataT> h_modelPDF(nElements, 0);
    std::vector<DataT> h_candidatePDF(nElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_real_distribution<DataT> realGenerator(0.0, 1.0);

    std::generate(h_modelPDF.begin(), h_modelPDF.end(),
                  [&]() { return realGenerator(dre); });
    std::generate(h_candidatePDF.begin(), h_candidatePDF.end(),
                  [&]() { return realGenerator(dre); });

    //allocating and initializing memory to the GPU
    CUDA_CHECK(hipStreamCreate(&stream));
    MLCommon::allocate(d_modelPDF, nElements, true);
    MLCommon::allocate(d_candidatePDF, nElements, true);

    MLCommon::updateDevice(d_modelPDF, &h_modelPDF[0], (int)nElements, stream);
    MLCommon::updateDevice(d_candidatePDF, &h_candidatePDF[0], (int)nElements,
                           stream);
    std::shared_ptr<MLCommon::deviceAllocator> allocator(
      new defaultDeviceAllocator);

    //generating the golden output
    for (int i = 0; i < nElements; ++i) {
      if (h_modelPDF[i] == 0.0)
        truthklDivergence += 0;

      else
        truthklDivergence +=
          h_modelPDF[i] * log(h_modelPDF[i] / h_candidatePDF[i]);
    }

    //calling the klDivergence CUDA implementation
    computedklDivergence = MLCommon::Metrics::klDivergence(
      d_modelPDF, d_candidatePDF, nElements, allocator, stream);
  }

  //the destructor
  void TearDown() override {
    CUDA_CHECK(hipFree(d_modelPDF));
    CUDA_CHECK(hipFree(d_candidatePDF));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  //declaring the data values
  klDivergenceParam params;
  DataT* d_modelPDF = nullptr;
  DataT* d_candidatePDF = nullptr;
  int nElements = 0;
  DataT truthklDivergence = 0;
  DataT computedklDivergence = 0;
  hipStream_t stream;
};

//setting test parameter values
const std::vector<klDivergenceParam> inputs = {
  {500, 0.000001}, {200, 0.001}, {5000, 0.000001}, {500000, 0.000001}

};

//writing the test suite
typedef klDivergenceTest<double> klDivergenceTestClass;
TEST_P(klDivergenceTestClass, Result) {
  ASSERT_NEAR(computedklDivergence, truthklDivergence, params.tolerance);
}
INSTANTIATE_TEST_CASE_P(klDivergence, klDivergenceTestClass,
                        ::testing::ValuesIn(inputs));

}  //end namespace Metrics
}  //end namespace MLCommon
