#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/subtract.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename Type>
__global__ void naiveSubtractElemKernel(Type *out, const Type *in1,
                                        const Type *in2, int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = in1[idx] - in2[idx];
  }
}

template <typename Type>
void naiveSubtractElem(Type *out, const Type *in1, const Type *in2, int len,
                       hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naiveSubtractElemKernel<Type><<<nblks, TPB, 0, stream>>>(out, in1, in2, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename Type>
__global__ void naiveSubtractScalarKernel(Type *out, const Type *in1,
                                          const Type in2, int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = in1[idx] - in2;
  }
}

template <typename Type>
void naiveSubtractScalar(Type *out, const Type *in1, const Type in2, int len,
                         hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naiveSubtractScalarKernel<Type>
    <<<nblks, TPB, 0, stream>>>(out, in1, in2, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct SubtractInputs {
  T tolerance;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const SubtractInputs<T> &dims) {
  return os;
}

template <typename T>
class SubtractTest : public ::testing::TestWithParam<SubtractInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<SubtractInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.len;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in1, len);
    allocate(in2, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in1, len, T(-1.0), T(1.0), stream);
    r.uniform(in2, len, T(-1.0), T(1.0), stream);

    naiveSubtractElem(out_ref, in1, in2, len, stream);
    naiveSubtractScalar(out_ref, out_ref, T(1), len, stream);

    subtract(out, in1, in2, len, stream);
    subtractScalar(out, out, T(1), len, stream);
    subtract(in1, in1, in2, len, stream);
    subtractScalar(in1, in1, T(1), len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

 protected:
  SubtractInputs<T> params;
  T *in1, *in2, *out_ref, *out;
};

const std::vector<SubtractInputs<float>> inputsf2 = {
  {0.000001f, 1024 * 1024, 1234ULL}};

const std::vector<SubtractInputs<double>> inputsd2 = {
  {0.00000001, 1024 * 1024, 1234ULL}};

typedef SubtractTest<float> SubtractTestF;
TEST_P(SubtractTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<float>(params.tolerance)));
}

typedef SubtractTest<double> SubtractTestD;
TEST_P(SubtractTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(SubtractTests, SubtractTestF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(SubtractTests, SubtractTestD,
                        ::testing::ValuesIn(inputsd2));

}  // end namespace LinAlg
}  // end namespace MLCommon
