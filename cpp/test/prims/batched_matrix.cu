#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <algorithm>
#include <cuml/common/cuml_allocator.hpp>
#include <cuml/cuml.hpp>
#include <random>
#include <vector>

#include "add.h"
#include "batched_matrix.h"
#include "matrix/batched_matrix.hpp"
#include "test_utils.h"

namespace MLCommon {
namespace Matrix {

enum BatchedMatrixOperation {
  AB_op,
  AZT_op,
  ZA_op,
  ApB_op,
  AmB_op,
  AkB_op,
  AsolveZ_op,
  LaggedZ_op,
  CopyA_op,
  CopyA2D_op,
  DiffA_op
};

template <typename T>
struct BatchedMatrixInputs {
  BatchedMatrixOperation operation;
  int n_batches;
  int m;  // Usually the dimensions of A and/or Z
  int n;
  int p;  // Usually the dimensions of B or other parameters
  int q;
  int s;  // Additional parameters for operations that need more than 4
  int t;
  T tolerance;
};

template <typename T>
class BatchedMatrixTest
  : public ::testing::TestWithParam<BatchedMatrixInputs<T>> {
 protected:
  void SetUp() override {
    using std::vector;
    params = ::testing::TestWithParam<BatchedMatrixInputs<T>>::GetParam();

    // Find out whether A, B and Z will be used (depending on the operation)
    bool use_A = (params.operation != LaggedZ_op);
    bool use_B = (params.operation == AB_op) || (params.operation == ApB_op) ||
                 (params.operation == AmB_op) || (params.operation == AkB_op);
    bool use_Z = (params.operation == AZT_op) || (params.operation == ZA_op) ||
                 (params.operation == AsolveZ_op) ||
                 (params.operation == LaggedZ_op);
    bool Z_col = (params.operation == AsolveZ_op);
    int r = params.operation == AZT_op ? params.n : params.m;

    // Check if the dimensions are valid and compute the output dimensions
    int m_r, n_r;
    switch (params.operation) {
      case AB_op:
        ASSERT_TRUE(params.n == params.p);
        m_r = params.m;
        n_r = params.q;
        break;
      case ApB_op:
      case AmB_op:
        ASSERT_TRUE(params.m == params.p && params.n == params.q);
        m_r = params.m;
        n_r = params.n;
        break;
      case AkB_op:
        m_r = params.m * params.p;
        n_r = params.n * params.q;
        break;
      case AZT_op:
        m_r = params.m;
        n_r = 1;
        break;
      case ZA_op:
        m_r = 1;
        n_r = params.n;
        break;
      case AsolveZ_op:
        ASSERT_TRUE(params.n == params.m);
        // For this test we multiply A by the solution and check against Z
        m_r = params.m;
        n_r = 1;
        break;
      case LaggedZ_op:
        // For this operation params.n holds the number of lags
        m_r = params.m - params.n;
        n_r = params.n;
        break;
      case CopyA_op:
        m_r = params.m;
        n_r = params.n;
        break;
      case CopyA2D_op:
        // For this operation p and q are the dimensions of the copy window
        m_r = params.p;
        n_r = params.q;
        break;
      case DiffA_op:
        // Note: A can represent either a row or column vector
        ASSERT_TRUE(params.m == 1 || params.n == 1);
        m_r = std::max(1, params.m - 1);
        n_r = std::max(1, params.n - 1);
    }

    // Create test matrices and vector
    std::vector<T> A;
    std::vector<T> B;
    std::vector<T> Z;
    if (use_A) A.resize(params.n_batches * params.m * params.n);
    if (use_B) B.resize(params.n_batches * params.p * params.q);
    if (use_Z) Z.resize(params.n_batches * r);

    // Generate random data
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<T> udis(-1.0, 3.0);
    for (int i = 0; i < A.size(); i++) A[i] = udis(gen);
    for (int i = 0; i < B.size(); i++) B[i] = udis(gen);
    for (int i = 0; i < Z.size(); i++) Z[i] = udis(gen);

    // Create handles, stream, allocator
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));
    auto allocator = std::make_shared<MLCommon::defaultDeviceAllocator>();

    // Created batched matrices
    BatchedMatrix<T> AbM(params.m, params.n, params.n_batches, handle,
                         allocator, stream);
    BatchedMatrix<T> BbM(params.p, params.q, params.n_batches, handle,
                         allocator, stream);
    BatchedMatrix<T> ZbM(Z_col ? r : 1, Z_col ? 1 : r, params.n_batches, handle,
                         allocator, stream);

    // Copy the data to the device
    if (use_A) updateDevice(AbM.raw_data(), A.data(), A.size(), stream);
    if (use_B) updateDevice(BbM.raw_data(), B.data(), B.size(), stream);
    if (use_Z) updateDevice(ZbM.raw_data(), Z.data(), Z.size(), stream);

    // Create fake batched matrices to be overwritten by results
    res_bM = new BatchedMatrix<T>(1, 1, 1, handle, allocator, stream);

    // Compute the tested results
    switch (params.operation) {
      case AB_op:
        *res_bM = AbM * BbM;
        break;
      case ApB_op:
        *res_bM = AbM + BbM;
        break;
      case AmB_op:
        *res_bM = AbM - BbM;
        break;
      case AkB_op:
        *res_bM = b_kron(AbM, BbM);
        break;
      case AZT_op:
        *res_bM = b_gemm(AbM, ZbM, false, true);
        break;
      case ZA_op:
        *res_bM = ZbM * AbM;
        break;
      case AsolveZ_op:
        // A * A\Z -> should be Z
        *res_bM = AbM * b_solve(AbM, ZbM);
        break;
      case LaggedZ_op:
        *res_bM = b_lagged_mat(ZbM, params.n);
        break;
      case CopyA_op:
        *res_bM = AbM.deepcopy();
        break;
      case CopyA2D_op:
        *res_bM = b_2dcopy(AbM, params.s, params.t, params.p, params.q);
        break;
      case DiffA_op:
        *res_bM = AbM.difference();
        break;
    }

    // Compute the expected results
    res_h.resize(params.n_batches * m_r * n_r);
    switch (params.operation) {
      case AB_op:
        for (int bid = 0; bid < params.n_batches; bid++) {
          naiveMatMul(res_h.data() + bid * m_r * n_r,
                      A.data() + bid * params.m * params.n,
                      B.data() + bid * params.p * params.q, params.m, params.n,
                      params.q);
        }
        break;
      case ApB_op:
        naiveAdd(res_h.data(), A.data(), B.data(), A.size());
        break;
      case AmB_op:
        naiveAdd(res_h.data(), A.data(), B.data(), A.size(), T(-1.0));
        break;
      case AkB_op:
        for (int bid = 0; bid < params.n_batches; bid++) {
          naiveKronecker(res_h.data() + bid * m_r * n_r,
                         A.data() + bid * params.m * params.n,
                         B.data() + bid * params.p * params.q, params.m,
                         params.n, params.p, params.q);
        }
        break;
      case AZT_op:
        for (int bid = 0; bid < params.n_batches; bid++) {
          naiveMatMul(res_h.data() + bid * m_r * n_r,
                      A.data() + bid * params.m * params.n, Z.data() + bid * r,
                      params.m, params.n, 1);
        }
        break;
      case ZA_op:
        for (int bid = 0; bid < params.n_batches; bid++) {
          naiveMatMul(res_h.data() + bid * m_r * n_r, Z.data() + bid * r,
                      A.data() + bid * params.m * params.n, 1, params.m,
                      params.n);
        }
        break;
      case AsolveZ_op:
        // Simply copy Z in the result
        memcpy(res_h.data(), Z.data(), r * params.n_batches * sizeof(T));
        break;
      case LaggedZ_op:
        for (int bid = 0; bid < params.n_batches; bid++) {
          naiveLaggedMat(res_h.data() + bid * m_r * n_r,
                         Z.data() + bid * params.m, params.m, params.n);
        }
        break;
      case CopyA_op:
        memcpy(res_h.data(), A.data(),
               params.m * params.n * params.n_batches * sizeof(T));
        break;
      case CopyA2D_op:
        for (int bid = 0; bid < params.n_batches; bid++) {
          naive2DCopy(res_h.data() + bid * m_r * n_r,
                      A.data() + bid * params.m * params.n, params.s, params.t,
                      params.m, m_r, n_r);
        }
        break;
      case DiffA_op:
        int len = params.m * params.n;
        for (int bid = 0; bid < params.n_batches; bid++) {
          naiveDiff(res_h.data() + bid * (len - 1), A.data() + bid * len, len);
        }
        break;
    }

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    delete res_bM;
    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  BatchedMatrixInputs<T> params;
  BatchedMatrix<T> *res_bM;
  std::vector<T> res_h;
  hipblasHandle_t handle;
  hipStream_t stream;
};

// Test parameters (op, n_batches, m, n, p, q, tolerance)
const std::vector<BatchedMatrixInputs<double>> inputsd = {
  {AB_op, 7, 15, 37, 37, 11, 0, 0, 1e-6},
  {AZT_op, 5, 33, 65, 1, 1, 0, 0, 1e-6},
  {ZA_op, 8, 12, 41, 1, 1, 0, 0, 1e-6},
  {ApB_op, 4, 16, 48, 16, 48, 0, 0, 1e-6},
  {AmB_op, 17, 9, 3, 9, 3, 0, 0, 1e-6},
  {AkB_op, 5, 3, 13, 31, 8, 0, 0, 1e-6},
  {AkB_op, 3, 7, 12, 31, 15, 0, 0, 1e-6},
  {AkB_op, 2, 11, 2, 8, 46, 0, 0, 1e-6},
  {AsolveZ_op, 6, 17, 17, 1, 1, 0, 0, 1e-6},
  {LaggedZ_op, 5, 31, 9, 1, 1, 0, 0, 1e-6},
  {LaggedZ_op, 7, 129, 3, 1, 1, 0, 0, 1e-6},
  {CopyA_op, 7, 35, 43, 1, 1, 0, 0, 1e-6},
  {CopyA2D_op, 11, 31, 63, 17, 14, 5, 9, 1e-6},
  {CopyA2D_op, 4, 33, 7, 30, 4, 3, 0, 1e-6},
  {DiffA_op, 5, 11, 1, 1, 1, 0, 0, 1e-6},
  {DiffA_op, 15, 1, 37, 1, 1, 0, 0, 1e-6}};

// Test parameters (op, n_batches, m, n, p, q, tolerance)
const std::vector<BatchedMatrixInputs<float>> inputsf = {
  {AB_op, 7, 15, 37, 37, 11, 0, 0, 1e-2},
  {AZT_op, 5, 33, 65, 1, 1, 0, 0, 1e-2},
  {ZA_op, 8, 12, 41, 1, 1, 0, 0, 1e-2},
  {ApB_op, 4, 16, 48, 16, 48, 0, 0, 1e-2},
  {AmB_op, 17, 9, 3, 9, 3, 0, 0, 1e-2},
  {AkB_op, 5, 3, 13, 31, 8, 0, 0, 1e-2},
  {AkB_op, 3, 7, 12, 31, 15, 0, 0, 1e-2},
  {AkB_op, 2, 11, 2, 8, 46, 0, 0, 1e-2},
  {AsolveZ_op, 6, 17, 17, 1, 1, 0, 0, 1e-2},
  {LaggedZ_op, 5, 31, 9, 1, 1, 0, 0, 1e-5},
  {LaggedZ_op, 7, 129, 3, 1, 1, 0, 0, 1e-5},
  {CopyA_op, 7, 35, 43, 1, 1, 0, 0, 1e-5},
  {CopyA2D_op, 11, 31, 63, 17, 14, 5, 9, 1e-5},
  {CopyA2D_op, 4, 33, 7, 30, 4, 3, 0, 1e-5},
  {DiffA_op, 5, 11, 1, 1, 1, 0, 0, 1e-2},
  {DiffA_op, 15, 1, 37, 1, 1, 0, 0, 1e-2}};

using BatchedMatrixTestD = BatchedMatrixTest<double>;
using BatchedMatrixTestF = BatchedMatrixTest<float>;
TEST_P(BatchedMatrixTestD, Result) {
  ASSERT_TRUE(devArrMatchHost(res_h.data(), res_bM->raw_data(), res_h.size(),
                              CompareApprox<double>(params.tolerance), stream));
}
TEST_P(BatchedMatrixTestF, Result) {
  ASSERT_TRUE(devArrMatchHost(res_h.data(), res_bM->raw_data(), res_h.size(),
                              CompareApprox<float>(params.tolerance), stream));
}

INSTANTIATE_TEST_CASE_P(BatchedMatrixTests, BatchedMatrixTestD,
                        ::testing::ValuesIn(inputsd));
INSTANTIATE_TEST_CASE_P(BatchedMatrixTests, BatchedMatrixTestF,
                        ::testing::ValuesIn(inputsf));

}  // namespace Matrix
}  // namespace MLCommon
