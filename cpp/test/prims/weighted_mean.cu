#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include "cuda_utils.h"
#include "random/rng.h"
#include "stats/weighted_mean.h"
#include "test_utils.h"

namespace MLCommon {
namespace Stats {

template <typename T>
struct WeightedMeanInputs {
  T tolerance;
  int M, N;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const WeightedMeanInputs<T>& I) {
  return os << "{ " << I.tolerance << ", " << I.M << ", " << I.N << ", "
            << I.seed << "}" << std::endl;
}

///// weighted row-wise mean test and support functions
template <typename T>
void naiveRowWeightedMean(T* R, T* D, T* W, int M, int N, bool rowMajor) {
  int istr = rowMajor ? 1 : M;
  int jstr = rowMajor ? N : 1;

  //sum the weights
  T WS = 0;
  for (int i = 0; i < N; i++) WS += W[i];

  for (int j = 0; j < M; j++) {
    R[j] = (T)0;
    for (int i = 0; i < N; i++) {
      //R[j] += (W[i]*D[i*istr + j*jstr] - R[j])/(T)(i+1);
      R[j] += (W[i] * D[i * istr + j * jstr]) / WS;
    }
  }
}

template <typename T>
class RowWeightedMeanTest
  : public ::testing::TestWithParam<WeightedMeanInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<WeightedMeanInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.M, cols = params.N, len = rows * cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    //device-side data
    din.resize(len);
    dweights.resize(cols);
    dexp.resize(rows);
    dact.resize(rows);

    //create random matrix and weights
    r.uniform(din.data().get(), len, T(-1.0), T(1.0), stream);
    r.uniform(dweights.data().get(), cols, T(-1.0), T(1.0), stream);

    //host-side data
    thrust::host_vector<T> hin = din;
    thrust::host_vector<T> hweights = dweights;
    thrust::host_vector<T> hexp(rows);

    //compute naive result & copy to GPU
    naiveRowWeightedMean(hexp.data(), hin.data(), hweights.data(), rows, cols,
                         true);
    dexp = hexp;

    //compute ml-prims result
    rowWeightedMean(dact.data().get(), din.data().get(), dweights.data().get(),
                    cols, rows, stream);

    //adjust tolerance to account for round-off accumulation
    params.tolerance *= params.N;
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {}

 protected:
  WeightedMeanInputs<T> params;
  thrust::host_vector<T> hin, hweights;
  thrust::device_vector<T> din, dweights, dexp, dact;
};

///// weighted column-wise mean test and support functions
template <typename T>
void naiveColWeightedMean(T* R, T* D, T* W, int M, int N, bool rowMajor) {
  int istr = rowMajor ? 1 : M;
  int jstr = rowMajor ? N : 1;

  //sum the weights
  T WS = 0;
  for (int j = 0; j < M; j++) WS += W[j];

  for (int i = 0; i < N; i++) {
    R[i] = (T)0;
    for (int j = 0; j < M; j++) {
      //R[i] += (W[j]*D[i*istr + j*jstr] - R[i])/(T)(j+1);
      R[i] += (W[j] * D[i * istr + j * jstr]) / WS;
    }
  }
}

template <typename T>
class ColWeightedMeanTest
  : public ::testing::TestWithParam<WeightedMeanInputs<T>> {
  void SetUp() override {
    params = ::testing::TestWithParam<WeightedMeanInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.M, cols = params.N, len = rows * cols;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    //device-side data
    din.resize(len);
    dweights.resize(rows);
    dexp.resize(cols);
    dact.resize(cols);

    //create random matrix and weights
    r.uniform(din.data().get(), len, T(-1.0), T(1.0), stream);
    r.uniform(dweights.data().get(), rows, T(-1.0), T(1.0), stream);

    //host-side data
    thrust::host_vector<T> hin = din;
    thrust::host_vector<T> hweights = dweights;
    thrust::host_vector<T> hexp(cols);

    //compute naive result & copy to GPU
    naiveColWeightedMean(hexp.data(), hin.data(), hweights.data(), rows, cols,
                         true);
    dexp = hexp;

    //compute ml-prims result
    colWeightedMean(dact.data().get(), din.data().get(), dweights.data().get(),
                    cols, rows, stream);

    //adjust tolerance to account for round-off accumulation
    params.tolerance *= params.M;
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {}

 protected:
  WeightedMeanInputs<T> params;
  thrust::host_vector<T> hin, hweights;
  thrust::device_vector<T> din, dweights, dexp, dact;
};

////// Parameter sets and test instantiation
static const float tolF = 128 * std::numeric_limits<float>::epsilon();
static const double tolD = 256 * std::numeric_limits<double>::epsilon();

const std::vector<WeightedMeanInputs<float>> inputsf = {
  {tolF, 4, 4, 1234},      {tolF, 1024, 32, 1234},  {tolF, 1024, 64, 1234},
  {tolF, 1024, 128, 1234}, {tolF, 1024, 256, 1234}, {tolF, 1024, 32, 1234},
  {tolF, 1024, 64, 1234},  {tolF, 1024, 128, 1234}, {tolF, 1024, 256, 1234}};

const std::vector<WeightedMeanInputs<double>> inputsd = {
  {tolD, 4, 4, 1234},      {tolD, 1024, 32, 1234},  {tolD, 1024, 64, 1234},
  {tolD, 1024, 128, 1234}, {tolD, 1024, 256, 1234}, {tolD, 1024, 32, 1234},
  {tolD, 1024, 64, 1234},  {tolD, 1024, 128, 1234}, {tolD, 1024, 256, 1234}};

using RowWeightedMeanTestF = RowWeightedMeanTest<float>;
TEST_P(RowWeightedMeanTestF, Result) {
  ASSERT_TRUE(devArrMatch(dexp.data().get(), dact.data().get(), params.M,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(RowWeightedMeanTest, RowWeightedMeanTestF,
                        ::testing::ValuesIn(inputsf));

using RowWeightedMeanTestD = RowWeightedMeanTest<double>;
TEST_P(RowWeightedMeanTestD, Result) {
  ASSERT_TRUE(devArrMatch(dexp.data().get(), dact.data().get(), params.M,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(RowWeightedMeanTest, RowWeightedMeanTestD,
                        ::testing::ValuesIn(inputsd));

using ColWeightedMeanTestF = ColWeightedMeanTest<float>;
TEST_P(ColWeightedMeanTestF, Result) {
  ASSERT_TRUE(devArrMatch(dexp.data().get(), dact.data().get(), params.N,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ColWeightedMeanTest, ColWeightedMeanTestF,
                        ::testing::ValuesIn(inputsf));

using ColWeightedMeanTestD = ColWeightedMeanTest<double>;
TEST_P(ColWeightedMeanTestD, Result) {
  ASSERT_TRUE(devArrMatch(dexp.data().get(), dact.data().get(), params.N,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ColWeightedMeanTest, ColWeightedMeanTestD,
                        ::testing::ValuesIn(inputsd));

};  // end namespace Stats
};  // end namespace MLCommon
