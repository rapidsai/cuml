#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <cache/cache.cuh>
#include <cuml/common/cuml_allocator.hpp>
#include <iostream>
#include <raft/cuda_utils.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Cache {

class CacheTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    allocator = std::shared_ptr<deviceAllocator>(
      new raft::mr::device::default_allocator());
    raft::allocate(x_dev, n_rows * n_cols);
    raft::update_device(x_dev, x_host, n_rows * n_cols, stream);
    raft::allocate(tile_dev, n_rows * n_cols);

    raft::allocate(keys_dev, n);
    raft::allocate(is_cached, n);
    raft::allocate(cache_idx_dev, n);
    raft::update_device(keys_dev, keys_host, n, stream);
    raft::allocate(zeroone_dev, n);
    raft::allocate(int_array_dev, 12);
    raft::update_device(zeroone_dev, zeroone_host, n, stream);
    raft::allocate(argfirst_dev, n_rows);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(tile_dev));
    CUDA_CHECK(hipFree(keys_dev));
    CUDA_CHECK(hipFree(cache_idx_dev));
    CUDA_CHECK(hipFree(is_cached));
    CUDA_CHECK(hipFree(zeroone_dev));
    CUDA_CHECK(hipFree(int_array_dev));
    CUDA_CHECK(hipFree(argfirst_dev));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  int n_rows = 10;
  int n_cols = 2;
  int n = 10;

  float *x_dev;
  int *keys_dev;
  int *cache_idx_dev;
  int *int_array_dev;
  float x_host[20] = {1,  2,  3,  4,  5,  6,  7,  8,  9,  10,
                      11, 12, 13, 14, 15, 16, 17, 18, 19, 20};

  float *tile_dev;

  int keys_host[10] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};

  int zeroone_host[10] = {0, 0, 0, 0, 0, 1, 1, 1, 1, 1};
  int *zeroone_dev;

  int *argfirst_dev;

  std::shared_ptr<deviceAllocator> allocator;
  hipStream_t stream;

  bool *is_cached;
};

__global__ void test_argfirst(const int *array, int n, int *res) {
  int k = threadIdx.x;
  res[k] = arg_first_ge(array, n, k);
}

TEST_F(CacheTest, TestArgFirst) {
  int argfirst_host[10] = {0, 1, 1, 1, 2, 2, 4, 4, 6, 7};
  raft::update_device(argfirst_dev, argfirst_host, 10, stream);

  test_argfirst<<<1, 10>>>(argfirst_dev, 10, int_array_dev);
  int idx_exp[10] = {0, 1, 4, 6, 6, 8, 8, 9, 10, 10};
  EXPECT_TRUE(
    devArrMatchHost(idx_exp, int_array_dev, 10, raft::Compare<int>()));
}

__global__ void test_nth_occurrence(const int *array, int n, int val,
                                    int *res) {
  int k = threadIdx.x;
  res[k] = find_nth_occurrence(array, n, val, k);
}

TEST_F(CacheTest, TestNthOccurrence) {
  test_nth_occurrence<<<1, 10>>>(zeroone_dev, 10, 0, int_array_dev);
  int idx_exp[10] = {0, 1, 2, 3, 4, -1, -1, -1, -1, -1};
  EXPECT_TRUE(
    devArrMatchHost(idx_exp, int_array_dev, 10, raft::Compare<int>()));
  test_nth_occurrence<<<1, 10>>>(zeroone_dev, 10, 1, int_array_dev);
  int idx_exp2[10] = {5, 6, 7, 8, 9, -1, -1, -1, -1, -1};
  EXPECT_TRUE(
    devArrMatchHost(idx_exp2, int_array_dev, 10, raft::Compare<int>()));
}

template <int nthreads, int associativity>
__global__ void test_rank_set_entries(const int *array, int n, int *res) {
  const int items_per_thread = raft::ceildiv(associativity, nthreads);
  __shared__ int rank[items_per_thread * nthreads];

  rank_set_entries<nthreads, associativity>(array, n, rank);

  int block_offset = blockIdx.x * associativity;

  for (int i = 0; i < items_per_thread; i++) {
    int k = threadIdx.x * items_per_thread + i;
    if (k < associativity && block_offset + k < n)
      res[block_offset + k] = rank[k];
  }
}

TEST_F(CacheTest, TestRankEntries) {
  // Three cache sets, with 4 elements each
  int val[12] = {12, 11, 10, 9, 8, 6, 7, 5, 4, 1, 2, 3};
  raft::update_device(int_array_dev, val, 12, stream);

  const int nthreads = 4;
  test_rank_set_entries<nthreads, 4>
    <<<3, nthreads>>>(int_array_dev, 12, int_array_dev);

  // expect that each block is sorted separately
  // the indices that sorts the block are the following
  int idx_exp[12] = {3, 2, 1, 0, 3, 1, 2, 0, 3, 0, 1, 2};

  EXPECT_TRUE(
    devArrMatchHost(idx_exp, int_array_dev, 12, raft::Compare<int>()));

  // do the same with less than 4 threads
  const int nthreads3 = 3;
  raft::update_device(int_array_dev, val, 12, stream);
  test_rank_set_entries<nthreads3, 4>
    <<<3, nthreads3>>>(int_array_dev, 12, int_array_dev);
  EXPECT_TRUE(
    devArrMatchHost(idx_exp, int_array_dev, 12, raft::Compare<int>()));
}

TEST_F(CacheTest, TestSimple) {
  float cache_size = 5 * sizeof(float) * n_cols / (1024 * 1024.0);
  Cache<float, 2> cache(allocator, stream, n_cols, cache_size);

  ASSERT_EQ(cache.GetSize(), 4);

  cache.GetCacheIdx(keys_dev, n, cache_idx_dev, is_cached, stream);
  EXPECT_TRUE(devArrMatch(false, is_cached, n, raft::Compare<bool>()));

  int cache_set[10] = {0, 1, 0, 1, 0, 1, 0, 1, 0, 1};
  EXPECT_TRUE(
    devArrMatchHost(cache_set, cache_idx_dev, n, raft::Compare<int>()));
  int n_cached = 1;
  cache.GetCacheIdxPartitioned(keys_dev, n, cache_idx_dev, &n_cached, stream);
  EXPECT_EQ(n_cached, 0);
}

TEST_F(CacheTest, TestAssignCacheIdx) {
  float cache_size = 5 * sizeof(float) * n_cols / (1024 * 1024.0);
  Cache<float, 2> cache(allocator, stream, n_cols, cache_size);

  ASSERT_EQ(cache.GetSize(), 4);

  int n_cached;
  cache.GetCacheIdxPartitioned(keys_dev, n, cache_idx_dev, &n_cached, stream);

  cache.AssignCacheIdx(keys_dev, n, cache_idx_dev, stream);

  int cache_idx_exp[10] = {0, 1, -1, -1, -1, 2, 3, -1, -1, -1};
  int keys_exp[10] = {8, 6, 4, 2, 0, 9, 7, 5, 3, 1};
  EXPECT_TRUE(
    devArrMatchHost(cache_idx_exp, cache_idx_dev, n, raft::Compare<int>()));
  EXPECT_TRUE(devArrMatchHost(keys_exp, keys_dev, n, raft::Compare<int>()));

  // Now the elements that have been assigned a cache slot are considered cached
  // A subsequent cache lookup should give us their cache indices.
  raft::update_device(keys_dev, keys_host, n, stream);
  cache.GetCacheIdxPartitioned(keys_dev, n, cache_idx_dev, &n_cached, stream);
  ASSERT_EQ(n_cached, 4);

  int keys_exp2[4] = {6, 7, 8, 9};
  EXPECT_TRUE(
    devArrMatchHost(keys_exp2, keys_dev, n_cached, raft::Compare<int>()));
  int cache_idx_exp2[4] = {1, 3, 0, 2};
  EXPECT_TRUE(devArrMatchHost(cache_idx_exp2, cache_idx_dev, n_cached,
                              raft::Compare<int>()));

  // Find cache slots, when not available
  int non_cached = n - n_cached;
  cache.AssignCacheIdx(keys_dev + n_cached, non_cached,
                       cache_idx_dev + n_cached, stream);

  int cache_idx_exp3[6] = {-1, -1, -1, -1, -1, -1};
  EXPECT_TRUE(devArrMatchHost(cache_idx_exp3, cache_idx_dev + n_cached,
                              non_cached, raft::Compare<int>()));
}

TEST_F(CacheTest, TestEvict) {
  float cache_size = 8 * sizeof(float) * n_cols / (1024 * 1024.0);
  Cache<float, 4> cache(allocator, stream, n_cols, cache_size);

  ASSERT_EQ(cache.GetSize(), 8);

  int n_cached;
  cache.GetCacheIdxPartitioned(keys_dev, 5, cache_idx_dev, &n_cached, stream);
  ASSERT_EQ(n_cached, 0);
  cache.AssignCacheIdx(keys_dev, 5, cache_idx_dev, stream);

  int cache_idx_exp[5] = {0, 1, 2, 4, 5};
  int keys_exp[5] = {4, 2, 0, 3, 1};
  EXPECT_TRUE(
    devArrMatchHost(cache_idx_exp, cache_idx_dev, 5, raft::Compare<int>()));
  EXPECT_TRUE(devArrMatchHost(keys_exp, keys_dev, 5, raft::Compare<int>()));

  int idx_host[10] = {2, 3, 4, 5, 6, 7, 8, 9, 10, 11};
  raft::update_device(keys_dev, idx_host, 10, stream);
  cache.GetCacheIdxPartitioned(keys_dev, 10, cache_idx_dev, &n_cached, stream);
  EXPECT_EQ(n_cached, 3);
  int cache_idx_exp2[3] = {1, 4, 0};
  EXPECT_TRUE(
    devArrMatchHost(cache_idx_exp2, cache_idx_dev, 3, raft::Compare<int>()));

  cache.AssignCacheIdx(keys_dev + n_cached, 10 - n_cached,
                       cache_idx_dev + n_cached, stream);

  int keys_exp3[10] = {2, 3, 4, 10, 8, 6, 11, 9, 7, 5};
  int cache_idx_exp3[10] = {1, 4, 0, 3, 2, -1, 6, 7, 5, -1};
  EXPECT_TRUE(devArrMatchHost(keys_exp3, keys_dev, 10, raft::Compare<int>()));
  EXPECT_TRUE(
    devArrMatchHost(cache_idx_exp3, cache_idx_dev, 10, raft::Compare<int>()));
}

TEST_F(CacheTest, TestStoreCollect) {
  float cache_size = 8 * sizeof(float) * n_cols / (1024 * 1024.0);
  Cache<float, 4> cache(allocator, stream, n_cols, cache_size);

  ASSERT_EQ(cache.GetSize(), 8);

  int n_cached;

  cache.GetCacheIdxPartitioned(keys_dev, 5, cache_idx_dev, &n_cached, stream);
  cache.AssignCacheIdx(keys_dev, 5, cache_idx_dev, stream);
  cache.GetCacheIdxPartitioned(keys_dev, 5, cache_idx_dev, &n_cached, stream);

  cache.StoreVecs(x_dev, 10, n_cached, cache_idx_dev, stream, keys_dev);
  cache.GetCacheIdxPartitioned(keys_dev, 5, cache_idx_dev, &n_cached, stream);
  cache.GetVecs(cache_idx_dev, n_cached, tile_dev, stream);

  int cache_idx_host[10];
  raft::update_host(cache_idx_host, cache_idx_dev, n_cached, stream);
  int keys_host[10];
  raft::update_host(keys_host, keys_dev, n_cached, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  for (int i = 0; i < n_cached; i++) {
    EXPECT_TRUE(devArrMatch(x_dev + keys_host[i] * n_cols,
                            tile_dev + i * n_cols, n_cols,
                            raft::Compare<int>()))
      << "vector " << i;
  }

  for (int k = 0; k < 4; k++) {
    cache.GetCacheIdxPartitioned(keys_dev, 10, cache_idx_dev, &n_cached,
                                 stream);
    if (k == 0) {
      EXPECT_EQ(n_cached, 5);
    } else {
      EXPECT_EQ(n_cached, 8);
    }

    cache.AssignCacheIdx(keys_dev + n_cached, 10 - n_cached,
                         cache_idx_dev + n_cached, stream);
    cache.StoreVecs(x_dev, 10, 10 - n_cached, cache_idx_dev + n_cached, stream,
                    keys_dev + n_cached);

    cache.GetVecs(cache_idx_dev, 10, tile_dev, stream);

    raft::update_host(cache_idx_host, cache_idx_dev, 10, stream);
    raft::update_host(keys_host, keys_dev, 10, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i = 0; i < 10; i++) {
      if (cache_idx_host[i] >= 0) {
        EXPECT_TRUE(devArrMatch(x_dev + keys_host[i] * n_cols,
                                tile_dev + i * n_cols, n_cols,
                                raft::Compare<int>()))
          << "vector " << i;
      }
    }
  }
}
};  // end namespace Cache
};  // end namespace MLCommon
