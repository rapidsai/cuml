#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "cuda_utils.h"
#include "distance/distance.h"
#include "random/rng.h"
#include "test_utils.h"


namespace MLCommon {
namespace Distance {

template <typename DataType>
__global__ void naiveDistanceAdjKernel(bool *dist, const DataType *x, const DataType *y,
                                       int m, int n, int k, DataType eps) {
  int midx = threadIdx.x + blockIdx.x * blockDim.x;
  int nidx = threadIdx.y + blockIdx.y * blockDim.y;
  if (midx >= m || nidx >= n)
    return;
  DataType acc = DataType(0);
  for (int i = 0; i < k; ++i) {
    auto diff = x[i + midx * k] - y[i + nidx * k];
    acc += diff * diff;
  }
  dist[midx * n + nidx] = acc <= eps;
}

template <typename DataType>
void naiveDistanceAdj(bool *dist, const DataType *x, const DataType *y, int m, int n,
                      int k, DataType eps) {
  static const dim3 TPB(16, 32, 1);
  dim3 nblks(ceildiv(m, (int)TPB.x), ceildiv(n, (int)TPB.y), 1);
  naiveDistanceAdjKernel<DataType><<<nblks, TPB>>>(dist, x, y, m, n, k, eps);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename DataType>
struct DistanceAdjInputs {
  DataType eps;
  int m, n, k;
  unsigned long long int seed;
};

template <typename DataType>
::std::ostream &operator<<(::std::ostream &os, const DistanceAdjInputs<DataType> &dims) {
  return os;
}

template <typename DataType>
class DistanceAdjTest : public ::testing::TestWithParam<DistanceAdjInputs<DataType>> {
public:
  void SetUp() override {
    params = ::testing::TestWithParam<DistanceAdjInputs<DataType>>::GetParam();
    Random::Rng r(params.seed);
    int m = params.m;
    int n = params.n;
    int k = params.k;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(x, m * k);
    allocate(y, n * k);
    allocate(dist_ref, m * n);
    allocate(dist, m * n);
    r.uniform(x, m * k, DataType(-1.0), DataType(1.0), stream);
    r.uniform(y, n * k, DataType(-1.0), DataType(1.0), stream);

    DataType threshold = params.eps;

    naiveDistanceAdj(dist_ref, x, y, m, n, k, threshold);
    char *workspace = nullptr;
    size_t worksize = getWorkspaceSize<EucExpandedL2, DataType, DataType, bool>(x, y, m, n, k);
    if (worksize != 0) {
      allocate(workspace, worksize);
    }

    typedef cutlass::Shape<8, 128, 128> OutputTile_t;

    auto fin_op = [threshold] __device__(DataType d_val, int g_d_idx) {
      return d_val <= threshold;
    };

    distance<EucExpandedL2, DataType, DataType, bool, OutputTile_t>(
      x, y, dist, m, n, k, workspace, worksize, fin_op, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(workspace));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(x));
    CUDA_CHECK(hipFree(y));
    CUDA_CHECK(hipFree(dist_ref));
    CUDA_CHECK(hipFree(dist));
  }

protected:
  DistanceAdjInputs<DataType> params;
  DataType *x, *y;
  bool *dist_ref, *dist;
};


const std::vector<DistanceAdjInputs<float>> inputsf = {
  {0.01f, 1024, 1024, 32, 1234ULL},
  {0.1f, 1024, 1024, 32, 1234ULL},
  {1.0f, 1024, 1024, 32, 1234ULL},
  {10.0f, 1024, 1024, 32, 1234ULL}};
typedef DistanceAdjTest<float> DistanceAdjTestF;
TEST_P(DistanceAdjTestF, Result) {
  ASSERT_TRUE(devArrMatch(dist_ref, dist, params.m, params.n, Compare<bool>()));
}
INSTANTIATE_TEST_CASE_P(DistanceAdjTests, DistanceAdjTestF,
                        ::testing::ValuesIn(inputsf));


const std::vector<DistanceAdjInputs<double>> inputsd = {
  {0.01, 1024, 1024, 32, 1234ULL},
  {0.1, 1024, 1024, 32, 1234ULL},
  {1.0, 1024, 1024, 32, 1234ULL},
  {10.0, 1024, 1024, 32, 1234ULL}};
typedef DistanceAdjTest<double> DistanceAdjTestD;
TEST_P(DistanceAdjTestD, Result) {
  ASSERT_TRUE(devArrMatch(dist_ref, dist, params.m, params.n, Compare<bool>()));
}
INSTANTIATE_TEST_CASE_P(DistanceAdjTests, DistanceAdjTestD,
                        ::testing::ValuesIn(inputsd));

} // end namespace DistanceAdj
} // end namespace MLCommon
