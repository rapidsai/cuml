/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "cuda_utils.h"
#include "linalg/eig.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct EigInputs {
  T tolerance;
  int len;
  int n_row;
  int n_col;
  unsigned long long int seed;
  int n;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const EigInputs<T> &dims) {
  return os;
}

template <typename T>
class EigTest : public ::testing::TestWithParam<EigInputs<T>> {
 protected:
  void SetUp() override {
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUDA_CHECK(hipStreamCreate(&stream));
    std::shared_ptr<deviceAllocator> allocator(new defaultDeviceAllocator);

    params = ::testing::TestWithParam<EigInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.len;

    allocate(cov_matrix, len);
    T cov_matrix_h[] = {1.0,  0.9, 0.81, 0.729, 0.9,   1.0,  0.9, 0.81,
                        0.81, 0.9, 1.0,  0.9,   0.729, 0.81, 0.9, 1.0};
    ASSERT(len == 16, "This test only works with 4x4 matrices!");
    updateDevice(cov_matrix, cov_matrix_h, len, stream);

    allocate(eig_vectors, len);
    allocate(eig_vals, params.n_col);
    allocate(eig_vectors_jacobi, len);
    allocate(eig_vals_jacobi, params.n_col);

    T eig_vectors_ref_h[] = {0.2790, -0.6498, 0.6498, -0.2789, -0.5123, 0.4874,
                             0.4874, -0.5123, 0.6498, 0.2789,  -0.2789, -0.6498,
                             0.4874, 0.5123,  0.5123, 0.4874};
    T eig_vals_ref_h[] = {0.0614, 0.1024, 0.3096, 3.5266};

    allocate(eig_vectors_ref, len);
    allocate(eig_vals_ref, params.n_col);

    updateDevice(eig_vectors_ref, eig_vectors_ref_h, len, stream);
    updateDevice(eig_vals_ref, eig_vals_ref_h, params.n_col, stream);

    eigDC(cov_matrix, params.n_row, params.n_col, eig_vectors, eig_vals,
          cusolverH, stream, allocator);

    T tol = 1.e-7;
    int sweeps = 15;
    eigJacobi(cov_matrix, params.n_row, params.n_col, eig_vectors_jacobi,
              eig_vals_jacobi, cusolverH, stream, allocator, tol, sweeps);

    // test code for comparing two methods
    len = params.n * params.n;
    allocate(cov_matrix_large, len);
    allocate(eig_vectors_large, len);
    allocate(eig_vectors_jacobi_large, len);
    allocate(eig_vals_large, params.n);
    allocate(eig_vals_jacobi_large, params.n);

    r.uniform(cov_matrix_large, len, T(-1.0), T(1.0), stream);

    eigDC(cov_matrix_large, params.n, params.n, eig_vectors_large,
          eig_vals_large, cusolverH, stream, allocator);
    eigJacobi(cov_matrix_large, params.n, params.n, eig_vectors_jacobi_large,
              eig_vals_jacobi_large, cusolverH, stream, allocator, tol, sweeps);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(cov_matrix));
    CUDA_CHECK(hipFree(eig_vectors));
    CUDA_CHECK(hipFree(eig_vectors_jacobi));
    CUDA_CHECK(hipFree(eig_vals));
    CUDA_CHECK(hipFree(eig_vals_jacobi));
    CUDA_CHECK(hipFree(eig_vectors_ref));
    CUDA_CHECK(hipFree(eig_vals_ref));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  EigInputs<T> params;
  T *cov_matrix, *eig_vectors, *eig_vectors_jacobi, *eig_vectors_ref, *eig_vals,
    *eig_vals_jacobi, *eig_vals_ref;

  T *cov_matrix_large, *eig_vectors_large, *eig_vectors_jacobi_large,
    *eig_vals_large, *eig_vals_jacobi_large;

  hipsolverHandle_t cusolverH = NULL;
  hipStream_t stream;
};

const std::vector<EigInputs<float>> inputsf2 = {
  {0.001f, 4 * 4, 4, 4, 1234ULL, 256}};

const std::vector<EigInputs<double>> inputsd2 = {
  {0.001, 4 * 4, 4, 4, 1234ULL, 256}};

typedef EigTest<float> EigTestValF;
TEST_P(EigTestValF, Result) {
  ASSERT_TRUE(devArrMatch(eig_vals_ref, eig_vals, params.n_col,
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef EigTest<double> EigTestValD;
TEST_P(EigTestValD, Result) {
  ASSERT_TRUE(devArrMatch(eig_vals_ref, eig_vals, params.n_col,
                          CompareApproxAbs<double>(params.tolerance)));
}

typedef EigTest<float> EigTestVecF;
TEST_P(EigTestVecF, Result) {
  ASSERT_TRUE(devArrMatch(eig_vectors_ref, eig_vectors, params.len,
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef EigTest<double> EigTestVecD;
TEST_P(EigTestVecD, Result) {
  ASSERT_TRUE(devArrMatch(eig_vectors_ref, eig_vectors, params.len,
                          CompareApproxAbs<double>(params.tolerance)));
}

typedef EigTest<float> EigTestValJacobiF;
TEST_P(EigTestValJacobiF, Result) {
  ASSERT_TRUE(devArrMatch(eig_vals_ref, eig_vals_jacobi, params.n_col,
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef EigTest<double> EigTestValJacobiD;
TEST_P(EigTestValJacobiD, Result) {
  ASSERT_TRUE(devArrMatch(eig_vals_ref, eig_vals_jacobi, params.n_col,
                          CompareApproxAbs<double>(params.tolerance)));
}

typedef EigTest<float> EigTestVecJacobiF;
TEST_P(EigTestVecJacobiF, Result) {
  ASSERT_TRUE(devArrMatch(eig_vectors_ref, eig_vectors_jacobi, params.len,
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef EigTest<double> EigTestVecJacobiD;
TEST_P(EigTestVecJacobiD, Result) {
  ASSERT_TRUE(devArrMatch(eig_vectors_ref, eig_vectors_jacobi, params.len,
                          CompareApproxAbs<double>(params.tolerance)));
}

typedef EigTest<float> EigTestVecCompareF;
TEST_P(EigTestVecCompareF, Result) {
  ASSERT_TRUE(devArrMatch(eig_vectors_large, eig_vectors_jacobi_large,
                          (params.n * params.n),
                          CompareApproxAbs<float>(params.tolerance)));
}

typedef EigTest<double> EigTestVecCompareD;
TEST_P(EigTestVecCompareD, Result) {
  ASSERT_TRUE(devArrMatch(eig_vectors_large, eig_vectors_jacobi_large,
                          (params.n * params.n),
                          CompareApproxAbs<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(EigTests, EigTestValF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(EigTests, EigTestValD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(EigTests, EigTestVecF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(EigTests, EigTestVecD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(EigTests, EigTestValJacobiF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(EigTests, EigTestValJacobiD,
                        ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(EigTests, EigTestVecJacobiF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(EigTests, EigTestVecJacobiD,
                        ::testing::ValuesIn(inputsd2));

}  // end namespace LinAlg
}  // end namespace MLCommon
