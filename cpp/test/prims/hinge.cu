/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <functions/hinge.cuh>
#include <random/rng.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Functions {

template <typename T>
struct HingeLossInputs {
  T tolerance;
  T n_rows;
  T n_cols;
  int len;
};

template <typename T>
class HingeLossTest : public ::testing::TestWithParam<HingeLossInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<HingeLossInputs<T>>::GetParam();
    int len = params.len;
    int n_rows = params.n_rows;
    int n_cols = params.n_cols;

    T *labels, *coef;

    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    allocator.reset(new raft::mr::device::default_allocator);

    allocate(in, len);
    allocate(out, 1);
    allocate(out_lasso, 1);
    allocate(out_ridge, 1);
    allocate(out_elasticnet, 1);
    allocate(out_grad, n_cols);
    allocate(out_lasso_grad, n_cols);
    allocate(out_ridge_grad, n_cols);
    allocate(out_elasticnet_grad, n_cols);
    allocate(out_ref, 1);
    allocate(out_lasso_ref, 1);
    allocate(out_ridge_ref, 1);
    allocate(out_elasticnet_ref, 1);
    allocate(out_grad_ref, n_cols);
    allocate(out_lasso_grad_ref, n_cols);
    allocate(out_ridge_grad_ref, n_cols);
    allocate(out_elasticnet_grad_ref, n_cols);

    allocate(labels, params.n_rows);
    allocate(coef, params.n_cols);

    T h_in[len] = {0.1, 0.35, -0.9, -1.4, 2.0, 3.1};
    updateDevice(in, h_in, len, stream);

    T h_labels[n_rows] = {0.3, 2.0, -1.1};
    updateDevice(labels, h_labels, n_rows, stream);

    T h_coef[n_cols] = {0.35, -0.24};
    updateDevice(coef, h_coef, n_cols, stream);

    T h_out_ref[1] = {2.6037};
    updateDevice(out_ref, h_out_ref, 1, stream);

    T h_out_lasso_ref[1] = {2.9577};
    updateDevice(out_lasso_ref, h_out_lasso_ref, 1, stream);

    T h_out_ridge_ref[1] = {2.71176};
    updateDevice(out_ridge_ref, h_out_ridge_ref, 1, stream);

    T h_out_elasticnet_ref[1] = {2.83473};
    updateDevice(out_elasticnet_ref, h_out_elasticnet_ref, 1, stream);

    T h_out_grad_ref[n_cols] = {-0.24333, -1.1933};
    updateDevice(out_grad_ref, h_out_grad_ref, n_cols, stream);

    T h_out_lasso_grad_ref[n_cols] = {0.3566, -1.7933};
    updateDevice(out_lasso_grad_ref, h_out_lasso_grad_ref, n_cols, stream);

    T h_out_ridge_grad_ref[n_cols] = {0.1766, -1.4813};
    updateDevice(out_ridge_grad_ref, h_out_ridge_grad_ref, n_cols, stream);

    T h_out_elasticnet_grad_ref[n_cols] = {0.2666, -1.63733};
    updateDevice(out_elasticnet_grad_ref, h_out_elasticnet_grad_ref, n_cols,
                 stream);

    T alpha = 0.6;
    T l1_ratio = 0.5;

    hingeLoss(in, params.n_rows, params.n_cols, labels, coef, out,
              penalty::NONE, alpha, l1_ratio, cublas_handle, allocator, stream);

    updateDevice(in, h_in, len, stream);

    hingeLossGrads(in, params.n_rows, params.n_cols, labels, coef, out_grad,
                   penalty::NONE, alpha, l1_ratio, cublas_handle, allocator,
                   stream);

    updateDevice(in, h_in, len, stream);

    hingeLoss(in, params.n_rows, params.n_cols, labels, coef, out_lasso,
              penalty::L1, alpha, l1_ratio, cublas_handle, allocator, stream);

    updateDevice(in, h_in, len, stream);

    hingeLossGrads(in, params.n_rows, params.n_cols, labels, coef,
                   out_lasso_grad, penalty::L1, alpha, l1_ratio, cublas_handle,
                   allocator, stream);

    updateDevice(in, h_in, len, stream);

    hingeLoss(in, params.n_rows, params.n_cols, labels, coef, out_ridge,
              penalty::L2, alpha, l1_ratio, cublas_handle, allocator, stream);

    hingeLossGrads(in, params.n_rows, params.n_cols, labels, coef,
                   out_ridge_grad, penalty::L2, alpha, l1_ratio, cublas_handle,
                   allocator, stream);

    updateDevice(in, h_in, len, stream);

    hingeLoss(in, params.n_rows, params.n_cols, labels, coef, out_elasticnet,
              penalty::ELASTICNET, alpha, l1_ratio, cublas_handle, allocator,
              stream);

    hingeLossGrads(in, params.n_rows, params.n_cols, labels, coef,
                   out_elasticnet_grad, penalty::ELASTICNET, alpha, l1_ratio,
                   cublas_handle, allocator, stream);

    updateDevice(in, h_in, len, stream);

    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(coef));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_lasso));
    CUDA_CHECK(hipFree(out_ridge));
    CUDA_CHECK(hipFree(out_elasticnet));
    CUDA_CHECK(hipFree(out_grad));
    CUDA_CHECK(hipFree(out_lasso_grad));
    CUDA_CHECK(hipFree(out_ridge_grad));
    CUDA_CHECK(hipFree(out_elasticnet_grad));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out_lasso_ref));
    CUDA_CHECK(hipFree(out_ridge_ref));
    CUDA_CHECK(hipFree(out_elasticnet_ref));
    CUDA_CHECK(hipFree(out_grad_ref));
    CUDA_CHECK(hipFree(out_lasso_grad_ref));
    CUDA_CHECK(hipFree(out_ridge_grad_ref));
    CUDA_CHECK(hipFree(out_elasticnet_grad_ref));
  }

 protected:
  HingeLossInputs<T> params;
  T *in;
  T *out, *out_lasso, *out_ridge, *out_elasticnet;
  T *out_ref, *out_lasso_ref, *out_ridge_ref, *out_elasticnet_ref;
  T *out_grad, *out_lasso_grad, *out_ridge_grad, *out_elasticnet_grad;
  T *out_grad_ref, *out_lasso_grad_ref, *out_ridge_grad_ref,
    *out_elasticnet_grad_ref;
  std::shared_ptr<deviceAllocator> allocator;
};

const std::vector<HingeLossInputs<float>> inputsf = {{0.01f, 3, 2, 6}};

const std::vector<HingeLossInputs<double>> inputsd = {{0.01, 3, 2, 6}};

typedef HingeLossTest<float> HingeLossTestF;
TEST_P(HingeLossTestF, Result) {
  ASSERT_TRUE(
    devArrMatch(out_ref, out, 1, CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_grad_ref, out_grad, params.n_cols,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.n_cols,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.n_cols,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.n_cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef HingeLossTest<double> HingeLossTestD;
TEST_P(HingeLossTestD, Result) {
  ASSERT_TRUE(
    devArrMatch(out_ref, out, 1, CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_grad_ref, out_grad, params.n_cols,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.n_cols,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.n_cols,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.n_cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(HingeLossTests, HingeLossTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(HingeLossTests, HingeLossTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Functions
}  // end namespace MLCommon
