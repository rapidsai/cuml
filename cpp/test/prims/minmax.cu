#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits>
#include <raft/cuda_utils.cuh>
#include <raft/random/rng.hpp>
#include <stats/minmax.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Stats {

///@todo: need to add tests for verifying the column subsampling feature

template <typename T>
struct MinMaxInputs {
  T tolerance;
  int rows, cols;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const MinMaxInputs<T>& dims)
{
  return os;
}

template <typename T>
__global__ void naiveMinMaxInitKernel(int ncols, T* globalmin, T* globalmax, T init_val)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= ncols) return;
  globalmin[tid] = init_val;
  globalmax[tid] = -init_val;
}

template <typename T>
__global__ void naiveMinMaxKernel(const T* data, int nrows, int ncols, T* globalmin, T* globalmax)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int col = tid / nrows;
  if (col < ncols) {
    T val = data[tid];
    if (!isnan(val)) {
      raft::myAtomicMin(&globalmin[col], val);
      raft::myAtomicMax(&globalmax[col], val);
    }
  }
}

template <typename T>
void naiveMinMax(
  const T* data, int nrows, int ncols, T* globalmin, T* globalmax, hipStream_t stream)
{
  const int TPB = 128;
  int nblks     = raft::ceildiv(ncols, TPB);
  T init_val    = std::numeric_limits<T>::max();
  naiveMinMaxInitKernel<<<nblks, TPB, 0, stream>>>(ncols, globalmin, globalmax, init_val);
  CUDA_CHECK(hipGetLastError());
  nblks = raft::ceildiv(nrows * ncols, TPB);
  naiveMinMaxKernel<<<nblks, TPB, 0, stream>>>(data, nrows, ncols, globalmin, globalmax);
  CUDA_CHECK(hipGetLastError());
}

template <typename T>
__global__ void nanKernel(T* data, const bool* mask, int len, T nan)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  if (!mask[tid]) data[tid] = nan;
}

template <typename T>
class MinMaxTest : public ::testing::TestWithParam<MinMaxInputs<T>> {
 protected:
  MinMaxTest() : minmax_act(0, stream), minmax_ref(0, stream) {}

  void SetUp() override
  {
    params = ::testing::TestWithParam<MinMaxInputs<T>>::GetParam();
    raft::random::Rng r(params.seed);
    int len = params.rows * params.cols;
    CUDA_CHECK(hipStreamCreate(&stream));

    rmm::device_uvector<T> data(len, stream);
    rmm::device_uvector<bool> mask(len, stream);
    minmax_act.resize(2 * params.cols, stream);
    minmax_ref.resize(2 * params.cols, stream);

    r.normal(data.data(), len, (T)0.0, (T)1.0, stream);
    T nan_prob = 0.01;
    r.bernoulli(mask.data(), len, nan_prob, stream);
    const int TPB = 256;
    nanKernel<<<raft::ceildiv(len, TPB), TPB, 0, stream>>>(
      data.data(), mask.data(), len, std::numeric_limits<T>::quiet_NaN());
    CUDA_CHECK(hipPeekAtLastError());
    naiveMinMax(data.data(),
                params.rows,
                params.cols,
                minmax_ref.data(),
                minmax_ref.data() + params.cols,
                stream);
    minmax<T, 512>(data.data(),
                   nullptr,
                   nullptr,
                   params.rows,
                   params.cols,
                   params.rows,
                   minmax_act.data(),
                   minmax_act.data() + params.cols,
                   nullptr,
                   stream);
  }

 protected:
  MinMaxInputs<T> params;
  rmm::device_uvector<T> minmax_act;
  rmm::device_uvector<T> minmax_ref;
  hipStream_t stream = 0;
};

const std::vector<MinMaxInputs<float>> inputsf = {{0.00001f, 1024, 32, 1234ULL},
                                                  {0.00001f, 1024, 64, 1234ULL},
                                                  {0.00001f, 1024, 128, 1234ULL},
                                                  {0.00001f, 1024, 256, 1234ULL},
                                                  {0.00001f, 1024, 512, 1234ULL},
                                                  {0.00001f, 1024, 1024, 1234ULL},
                                                  {0.00001f, 4096, 32, 1234ULL},
                                                  {0.00001f, 4096, 64, 1234ULL},
                                                  {0.00001f, 4096, 128, 1234ULL},
                                                  {0.00001f, 4096, 256, 1234ULL},
                                                  {0.00001f, 4096, 512, 1234ULL},
                                                  {0.00001f, 4096, 1024, 1234ULL},
                                                  {0.00001f, 8192, 32, 1234ULL},
                                                  {0.00001f, 8192, 64, 1234ULL},
                                                  {0.00001f, 8192, 128, 1234ULL},
                                                  {0.00001f, 8192, 256, 1234ULL},
                                                  {0.00001f, 8192, 512, 1234ULL},
                                                  {0.00001f, 8192, 1024, 1234ULL},
                                                  {0.00001f, 1024, 8192, 1234ULL}};

const std::vector<MinMaxInputs<double>> inputsd = {{0.0000001, 1024, 32, 1234ULL},
                                                   {0.0000001, 1024, 64, 1234ULL},
                                                   {0.0000001, 1024, 128, 1234ULL},
                                                   {0.0000001, 1024, 256, 1234ULL},
                                                   {0.0000001, 1024, 512, 1234ULL},
                                                   {0.0000001, 1024, 1024, 1234ULL},
                                                   {0.0000001, 4096, 32, 1234ULL},
                                                   {0.0000001, 4096, 64, 1234ULL},
                                                   {0.0000001, 4096, 128, 1234ULL},
                                                   {0.0000001, 4096, 256, 1234ULL},
                                                   {0.0000001, 4096, 512, 1234ULL},
                                                   {0.0000001, 4096, 1024, 1234ULL},
                                                   {0.0000001, 8192, 32, 1234ULL},
                                                   {0.0000001, 8192, 64, 1234ULL},
                                                   {0.0000001, 8192, 128, 1234ULL},
                                                   {0.0000001, 8192, 256, 1234ULL},
                                                   {0.0000001, 8192, 512, 1234ULL},
                                                   {0.0000001, 8192, 1024, 1234ULL},
                                                   {0.0000001, 1024, 8192, 1234ULL}};

typedef MinMaxTest<float> MinMaxTestF;
TEST_P(MinMaxTestF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(minmax_ref.data(),
                                minmax_act.data(),
                                2 * params.cols,
                                raft::CompareApprox<float>(params.tolerance)));
}

typedef MinMaxTest<double> MinMaxTestD;
TEST_P(MinMaxTestD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(minmax_ref.data(),
                                minmax_act.data(),
                                2 * params.cols,
                                raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(MinMaxTests, MinMaxTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(MinMaxTests, MinMaxTestD, ::testing::ValuesIn(inputsd));

}  // end namespace Stats
}  // end namespace MLCommon
