/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <matrix/gather.cuh>
#include <raft/cuda_utils.cuh>
#include <raft/random/rng.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Matrix {

template <typename MatrixIteratorT, typename MapIteratorT>
void naiveGatherImpl(MatrixIteratorT in, int D, int N, MapIteratorT map,
                     int map_length, MatrixIteratorT out) {
  for (int outRow = 0; outRow < map_length; ++outRow) {
    typename std::iterator_traits<MapIteratorT>::value_type map_val =
      map[outRow];
    int inRowStart = map_val * D;
    int outRowStart = outRow * D;
    for (int i = 0; i < D; ++i) {
      out[outRowStart + i] = in[inRowStart + i];
    }
  }
}

template <typename MatrixIteratorT, typename MapIteratorT>
void naiveGather(MatrixIteratorT in, int D, int N, MapIteratorT map,
                 int map_length, MatrixIteratorT out) {
  naiveGatherImpl(in, D, N, map, map_length, out);
}

template <typename MatrixIteratorT, typename MapIteratorT>
void gatherLaunch(MatrixIteratorT in, int D, int N, MapIteratorT map,
                  int map_length, MatrixIteratorT out, hipStream_t stream) {
  typedef typename std::iterator_traits<MapIteratorT>::value_type MapValueT;
  Matrix::gather(in, D, N, map, map_length, out, stream);
}

struct GatherInputs {
  uint32_t nrows;
  uint32_t ncols;
  uint32_t map_length;
  unsigned long long int seed;
};

template <typename MatrixT, typename MapT>
class GatherTest : public ::testing::TestWithParam<GatherInputs> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<GatherInputs>::GetParam();
    raft::random::Rng r(params.seed);
    raft::random::Rng r_int(params.seed);
    CUDA_CHECK(hipStreamCreate(&stream));

    uint32_t nrows = params.nrows;
    uint32_t ncols = params.ncols;
    uint32_t map_length = params.map_length;
    uint32_t len = nrows * ncols;

    // input matrix setup
    raft::allocate(d_in, nrows * ncols);
    h_in = (MatrixT *)malloc(sizeof(MatrixT) * nrows * ncols);
    r.uniform(d_in, len, MatrixT(-1.0), MatrixT(1.0), stream);
    raft::update_host(h_in, d_in, len, stream);

    // map setup
    raft::allocate(d_map, map_length);
    h_map = (MapT *)malloc(sizeof(MapT) * map_length);
    r_int.uniformInt(d_map, map_length, (MapT)0, nrows, stream);
    raft::update_host(h_map, d_map, map_length, stream);

    // expected and actual output matrix setup
    h_out = (MatrixT *)malloc(sizeof(MatrixT) * map_length * ncols);
    raft::allocate(d_out_exp, map_length * ncols);
    raft::allocate(d_out_act, map_length * ncols);

    // launch gather on the host and copy the results to device
    naiveGather(h_in, ncols, nrows, h_map, map_length, h_out);
    raft::update_device(d_out_exp, h_out, map_length * ncols, stream);

    // launch device version of the kernel
    gatherLaunch(d_in, ncols, nrows, d_map, map_length, d_out_act, stream);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }
  void TearDown() override {
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_map));
    CUDA_CHECK(hipFree(d_out_act));
    CUDA_CHECK(hipFree(d_out_exp));

    free(h_in);
    free(h_map);
    free(h_out);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  hipStream_t stream;
  GatherInputs params;
  MatrixT *d_in, *h_in, *d_out_exp, *d_out_act, *h_out;
  MapT *d_map, *h_map;
};

const std::vector<GatherInputs> inputs = {
  {1024, 32, 128, 1234ULL},  {1024, 32, 256, 1234ULL},
  {1024, 32, 512, 1234ULL},  {1024, 32, 1024, 1234ULL},
  {1024, 64, 128, 1234ULL},  {1024, 64, 256, 1234ULL},
  {1024, 64, 512, 1234ULL},  {1024, 64, 1024, 1234ULL},
  {1024, 128, 128, 1234ULL}, {1024, 128, 256, 1234ULL},
  {1024, 128, 512, 1234ULL}, {1024, 128, 1024, 1234ULL}};

typedef GatherTest<float, uint32_t> GatherTestF;
TEST_P(GatherTestF, Result) {
  ASSERT_TRUE(devArrMatch(d_out_exp, d_out_act,
                          params.map_length * params.ncols,
                          raft::Compare<float>()));
}

typedef GatherTest<double, uint32_t> GatherTestD;
TEST_P(GatherTestD, Result) {
  ASSERT_TRUE(devArrMatch(d_out_exp, d_out_act,
                          params.map_length * params.ncols,
                          raft::Compare<double>()));
}

INSTANTIATE_TEST_CASE_P(GatherTests, GatherTestF, ::testing::ValuesIn(inputs));
INSTANTIATE_TEST_CASE_P(GatherTests, GatherTestD, ::testing::ValuesIn(inputs));

}  // end namespace Matrix
}  // end namespace MLCommon
