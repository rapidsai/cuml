#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <algorithm>
#include <cuda_utils.cuh>
#include <cuml/common/cuml_allocator.hpp>
#include <iostream>
#include <metrics/entropy.cuh>
#include <random>
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

struct entropyParam {
  int nElements;
  int lowerLabelRange;
  int upperLabelRange;
  double tolerance;
};

//test fixture class
template <typename T>
class entropyTest : public ::testing::TestWithParam<entropyParam> {
 protected:
  //the constructor
  void SetUp() override {
    //getting the parameters
    params = ::testing::TestWithParam<entropyParam>::GetParam();

    nElements = params.nElements;
    lowerLabelRange = params.lowerLabelRange;
    upperLabelRange = params.upperLabelRange;

    //generating random value test input
    std::vector<int> arr1(nElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_int_distribution<int> intGenerator(lowerLabelRange,
                                                    upperLabelRange);

    std::generate(arr1.begin(), arr1.end(),
                  [&]() { return intGenerator(dre); });

    //generating the golden output
    int numUniqueClasses = upperLabelRange - lowerLabelRange + 1;

    int *p = (int *)malloc(numUniqueClasses * sizeof(int));
    memset(p, 0, numUniqueClasses * sizeof(int));

    //calculating the bincount array
    for (int i = 0; i < nElements; ++i) {
      ++p[arr1[i] - lowerLabelRange];
    }

    //calculating the aggregate entropy
    for (int i = 0; i < numUniqueClasses; ++i) {
      if (p[i])
        truthEntropy += -1 * (double(p[i]) / double(nElements)) *
                        (log(double(p[i])) - log(double(nElements)));
    }

    //allocating and initializing memory to the GPU
    CUDA_CHECK(hipStreamCreate(&stream));
    MLCommon::allocate(clusterArray, nElements, true);
    MLCommon::updateDevice(clusterArray, &arr1[0], (int)nElements, stream);

    std::shared_ptr<MLCommon::deviceAllocator> allocator(
      new raft::mr::device::default_allocator);

    CUDA_CHECK(hipStreamSynchronize(stream));
    //calling the entropy CUDA implementation
    computedEntropy =
      MLCommon::Metrics::entropy(clusterArray, nElements, lowerLabelRange,
                                 upperLabelRange, allocator, stream);
  }

  //the destructor
  void TearDown() override {
    CUDA_CHECK(hipFree(clusterArray));

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  //declaring the data values
  entropyParam params;
  T lowerLabelRange, upperLabelRange;
  T *clusterArray = nullptr;

  int nElements = 0;
  double truthEntropy = 0;
  double computedEntropy = 0;
  hipStream_t stream;
};

//setting test parameter values
const std::vector<entropyParam> inputs = {
  {199, 1, 10, 0.000001}, {200, 15, 100, 0.000001}, {100, 1, 20, 0.000001},
  {10, 1, 10, 0.000001},  {198, 1, 100, 0.000001},  {300, 3, 99, 0.000001},
  {199, 1, 10, 0.000001}, {200, 15, 100, 0.000001}, {100, 1, 20, 0.000001},
  {10, 1, 10, 0.000001},  {198, 1, 100, 0.000001},  {300, 3, 99, 0.000001}};

//writing the test suite
typedef entropyTest<int> entropyTestClass;
TEST_P(entropyTestClass, Result) {
  ASSERT_NEAR(computedEntropy, truthEntropy, params.tolerance);
}
INSTANTIATE_TEST_CASE_P(entropy, entropyTestClass, ::testing::ValuesIn(inputs));

}  //end namespace Metrics
}  //end namespace MLCommon
