#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "cuda_utils.h"
#include "decoupled_lookback.h"
#include "test_utils.h"

namespace MLCommon {

template <int TPB>
__global__ void dlbTestKernel(void *workspace, int len, int *out) {
  DecoupledLookBack<int> dlb(workspace);
  int count = threadIdx.x == blockDim.x - 1 ? 1 : 0;
  auto prefix = dlb(count);
  if (threadIdx.x == blockDim.x - 1) out[blockIdx.x] = prefix;
}

void dlbTest(int len, int *out) {
  constexpr int TPB = 256;
  int nblks = len;
  size_t workspaceSize = DecoupledLookBack<int>::computeWorkspaceSize(nblks);
  char *workspace;
  allocate(workspace, workspaceSize);
  CUDA_CHECK(hipMemset(workspace, 0, workspaceSize));
  dlbTestKernel<TPB><<<nblks, TPB>>>(workspace, len, out);
  CUDA_CHECK(hipPeekAtLastError());
  CUDA_CHECK(hipFree(workspace));
}

struct DlbInputs {
  int len;
};

::std::ostream &operator<<(::std::ostream &os, const DlbInputs &dims) {
  return os;
}

class DlbTest : public ::testing::TestWithParam<DlbInputs> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<DlbInputs>::GetParam();
    int len = params.len;
    allocate(out, len);
    dlbTest(len, out);
  }

  void TearDown() override { CUDA_CHECK(hipFree(out)); }

 protected:
  DlbInputs params;
  int *out;
};

template <typename T, typename L>
::testing::AssertionResult devArrMatchCustom(const T *actual, size_t size,
                                             L eq_compare,
                                             hipStream_t stream = 0) {
  std::vector<T> act_h(size);
  updateHost<T>(&(act_h[0]), actual, size, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
  for (size_t i(0); i < size; ++i) {
    auto act = act_h[i];
    auto expected = (T)i;
    if (!eq_compare(expected, act)) {
      return ::testing::AssertionFailure()
             << "actual=" << act << " != expected=" << expected << " @" << i;
    }
  }
  return ::testing::AssertionSuccess();
}

const std::vector<DlbInputs> inputs = {{4}, {16}, {64}, {256}, {2048}};
TEST_P(DlbTest, Result) {
  ASSERT_TRUE(devArrMatchCustom(out, params.len, Compare<int>()));
}
INSTANTIATE_TEST_CASE_P(DlbTests, DlbTest, ::testing::ValuesIn(inputs));

}  // end namespace MLCommon
