/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "matrix/matrix.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace Matrix {

template <typename T>
struct MatrixInputs {
  T tolerance;
  int n_row;
  int n_col;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const MatrixInputs<T> &dims) {
  return os;
}

template <typename T>
class MatrixTest : public ::testing::TestWithParam<MatrixInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MatrixInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.n_row * params.n_col;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in1, len);
    allocate(in2, len);
    allocate(in1_revr, len);
    r.uniform(in1, len, T(-1.0), T(1.0), stream);

    copy(in1, in2, params.n_row, params.n_col, stream);
    // copy(in1, in1_revr, params.n_row, params.n_col);
    // colReverse(in1_revr, params.n_row, params.n_col);

    T *outTrunc;
    allocate(outTrunc, 6);
    truncZeroOrigin(in1, params.n_row, outTrunc, 3, 2, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    // CUDA_CHECK(hipFree(in1_revr));
  }

 protected:
  MatrixInputs<T> params;
  T *in1, *in2, *in1_revr;
};

const std::vector<MatrixInputs<float>> inputsf2 = {{0.000001f, 4, 4, 1234ULL}};

const std::vector<MatrixInputs<double>> inputsd2 = {
  {0.00000001, 4, 4, 1234ULL}};

typedef MatrixTest<float> MatrixTestF;
TEST_P(MatrixTestF, Result) {
  ASSERT_TRUE(devArrMatch(in1, in2, params.n_row * params.n_col,
                          CompareApprox<float>(params.tolerance)));
}

typedef MatrixTest<double> MatrixTestD;
TEST_P(MatrixTestD, Result) {
  ASSERT_TRUE(devArrMatch(in1, in2, params.n_row * params.n_col,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(MatrixTests, MatrixTestF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(MatrixTests, MatrixTestD,
                        ::testing::ValuesIn(inputsd2));

}  // end namespace Matrix
}  // end namespace MLCommon
