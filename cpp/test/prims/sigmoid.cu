/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <functions/sigmoid.cuh>
#include <raft/cuda_utils.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Functions {

template <typename T>
struct SigmoidInputs {
  T tolerance;
  int len;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const SigmoidInputs<T>& dims) {
  return os;
}

template <typename T>
class SigmoidTest : public ::testing::TestWithParam<SigmoidInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<SigmoidInputs<T>>::GetParam();

    int len = params.len;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    raft::allocate(data, len);
    T data_h[params.len] = {2.1, -4.5, -0.34, 10.0};
    raft::update_device(data, data_h, len, stream);

    raft::allocate(result, len);
    raft::allocate(result_ref, len);
    T result_ref_h[params.len] = {0.89090318, 0.01098694, 0.41580948,
                                  0.9999546};
    raft::update_device(result_ref, result_ref_h, len, stream);

    sigmoid(result, data, len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(result));
    CUDA_CHECK(hipFree(result_ref));
  }

 protected:
  SigmoidInputs<T> params;
  T *data, *result, *result_ref;
};

const std::vector<SigmoidInputs<float>> inputsf2 = {{0.001f, 4}};

const std::vector<SigmoidInputs<double>> inputsd2 = {{0.001, 4}};

typedef SigmoidTest<float> SigmoidTestValF;
TEST_P(SigmoidTestValF, Result) {
  ASSERT_TRUE(
    raft::devArrMatch(result_ref, result, params.len,
                      raft::CompareApproxAbs<float>(params.tolerance)));
}

typedef SigmoidTest<double> SigmoidTestValD;
TEST_P(SigmoidTestValD, Result) {
  ASSERT_TRUE(
    raft::devArrMatch(result_ref, result, params.len,
                      raft::CompareApproxAbs<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(SigmoidTests, SigmoidTestValF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(SigmoidTests, SigmoidTestValD,
                        ::testing::ValuesIn(inputsd2));

}  // end namespace Functions
}  // end namespace MLCommon
