#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/power.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename Type>
__global__ void naivePowerElemKernel(Type *out, const Type *in1,
                                     const Type *in2, int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = myPow(in1[idx], in2[idx]);
  }
}

template <typename Type>
void naivePowerElem(Type *out, const Type *in1, const Type *in2, int len,
                    hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naivePowerElemKernel<Type><<<nblks, TPB, 0, stream>>>(out, in1, in2, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename Type>
__global__ void naivePowerScalarKernel(Type *out, const Type *in1,
                                       const Type in2, int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = myPow(in1[idx], in2);
  }
}

template <typename Type>
void naivePowerScalar(Type *out, const Type *in1, const Type in2, int len,
                      hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naivePowerScalarKernel<Type><<<nblks, TPB, 0, stream>>>(out, in1, in2, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct PowerInputs {
  T tolerance;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const PowerInputs<T> &dims) {
  return os;
}

template <typename T>
class PowerTest : public ::testing::TestWithParam<PowerInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<PowerInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.len;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in1, len);
    allocate(in2, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in1, len, T(1.0), T(2.0), stream);
    r.uniform(in2, len, T(1.0), T(2.0), stream);

    naivePowerElem(out_ref, in1, in2, len, stream);
    naivePowerScalar(out_ref, out_ref, T(2), len, stream);

    power(out, in1, in2, len, stream);
    powerScalar(out, out, T(2), len, stream);
    power(in1, in1, in2, len, stream);
    powerScalar(in1, in1, T(2), len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

 protected:
  PowerInputs<T> params;
  T *in1, *in2, *out_ref, *out;
  int device_count = 0;
};

const std::vector<PowerInputs<float>> inputsf2 = {
  {0.000001f, 1024 * 1024, 1234ULL}};

const std::vector<PowerInputs<double>> inputsd2 = {
  {0.00000001, 1024 * 1024, 1234ULL}};

typedef PowerTest<float> PowerTestF;
TEST_P(PowerTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<float>(params.tolerance)));
}

typedef PowerTest<double> PowerTestD;
TEST_P(PowerTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(PowerTests, PowerTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(PowerTests, PowerTestD, ::testing::ValuesIn(inputsd2));

}  // end namespace LinAlg
}  // end namespace MLCommon
