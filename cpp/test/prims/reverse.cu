/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <matrix/reverse.cuh>
#include <raft/random/rng.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Matrix {

template <typename T>
struct ReverseInputs {
  T tolerance;
  int nrows, ncols;
  bool rowMajor, alongRows;
  unsigned long long seed;
};

template <typename T>
class ReverseTest : public ::testing::TestWithParam<ReverseInputs<T>> {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    params = ::testing::TestWithParam<ReverseInputs<T>>::GetParam();
    raft::random::Rng r(params.seed);
    int len = params.nrows * params.ncols;
    raft::allocate(in, len);
    raft::allocate(out, len);
    r.uniform(in, len, T(-1.0), T(1.0), stream);
    // applying reverse twice should yield the same output!
    // this will in turn also verify the inplace mode of reverse method
    reverse(out, in, params.nrows, params.ncols, params.rowMajor,
            params.alongRows, stream);
    reverse(out, out, params.nrows, params.ncols, params.rowMajor,
            params.alongRows, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  ReverseInputs<T> params;
  T *in, *out;
  hipStream_t stream;
};

const std::vector<ReverseInputs<float>> inputsf = {
  {0.000001f, 32, 32, false, false, 1234ULL},
  {0.000001f, 32, 32, false, true, 1234ULL},
  {0.000001f, 32, 32, true, false, 1234ULL},
  {0.000001f, 32, 32, true, true, 1234ULL},

  {0.000001f, 41, 41, false, false, 1234ULL},
  {0.000001f, 41, 41, false, true, 1234ULL},
  {0.000001f, 41, 41, true, false, 1234ULL},
  {0.000001f, 41, 41, true, true, 1234ULL}};
typedef ReverseTest<float> ReverseTestF;
TEST_P(ReverseTestF, Result) {
  ASSERT_TRUE(devArrMatch(in, out, params.nrows, params.ncols,
                          raft::CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReverseTests, ReverseTestF,
                        ::testing::ValuesIn(inputsf));

typedef ReverseTest<double> ReverseTestD;
const std::vector<ReverseInputs<double>> inputsd = {
  {0.000001, 32, 32, false, false, 1234ULL},
  {0.000001, 32, 32, false, true, 1234ULL},
  {0.000001, 32, 32, true, false, 1234ULL},
  {0.000001, 32, 32, true, true, 1234ULL},

  {0.000001, 41, 41, false, false, 1234ULL},
  {0.000001, 41, 41, false, true, 1234ULL},
  {0.000001, 41, 41, true, false, 1234ULL},
  {0.000001, 41, 41, true, true, 1234ULL}};
TEST_P(ReverseTestD, Result) {
  ASSERT_TRUE(devArrMatch(in, out, params.nrows, params.ncols,
                          raft::CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReverseTests, ReverseTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Matrix
}  // end namespace MLCommon
