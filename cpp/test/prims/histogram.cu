#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <random/rng.h>
#include <stats/histogram.h>
#include "test_utils.h"

namespace MLCommon {
namespace Stats {

// Note: this kernel also updates the input vector to take care of OOB bins!
__global__ void naiveHistKernel(int* bins, int nbins, int* in, int nrows) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  auto offset = blockIdx.y * nrows;
  auto binOffset = blockIdx.y * nbins;
  for (; tid < nrows; tid += stride) {
    int id = in[offset + tid];
    if (id < 0)
      id = 0;
    else if (id >= nbins)
      id = nbins - 1;
    in[offset + tid] = id;
    atomicAdd(bins + binOffset + id, 1);
  }
}

void naiveHist(int* bins, int nbins, int* in, int nrows, int ncols,
               hipStream_t stream) {
  const int TPB = 128;
  int nblksx = ceildiv(nrows, TPB);
  dim3 blks(nblksx, ncols);
  naiveHistKernel<<<blks, TPB, 0, stream>>>(bins, nbins, in, nrows);
  CUDA_CHECK(hipGetLastError());
}

struct HistInputs {
  int nrows, ncols, nbins;
  bool isNormal;
  HistType type;
  int start, end;
  unsigned long long int seed;
};

class HistTest : public ::testing::TestWithParam<HistInputs> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<HistInputs>::GetParam();
    Random::Rng r(params.seed);
    CUDA_CHECK(hipStreamCreate(&stream));
    int len = params.nrows * params.ncols;
    allocate(in, len);
    if (params.isNormal) {
      r.normalInt(in, len, params.start, params.end, stream);
    } else {
      r.uniformInt(in, len, params.start, params.end, stream);
    }
    allocate(bins, params.nbins * params.ncols);
    allocate(ref_bins, params.nbins * params.ncols);
    CUDA_CHECK(hipMemsetAsync(
      ref_bins, 0, sizeof(int) * params.nbins * params.ncols, stream));
    naiveHist(ref_bins, params.nbins, in, params.nrows, params.ncols, stream);
    histogram<int>(params.type, bins, params.nbins, in, params.nrows,
                   params.ncols, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(bins));
    CUDA_CHECK(hipFree(ref_bins));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  hipStream_t stream;
  HistInputs params;
  int* in;
  int *bins, *ref_bins;
};

static const int oneK = 1024;
static const int oneM = oneK * oneK;
const std::vector<HistInputs> inputs = {
  {oneM, 1, 2 * oneM, false, HistTypeGmem, 0, 2 * oneM, 1234ULL},
  {oneM, 1, 2 * oneM, true, HistTypeGmem, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneM, false, HistTypeGmem, 0, 2 * oneM, 1234ULL},
  {oneM + 1, 1, 2 * oneM, true, HistTypeGmem, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneM, false, HistTypeGmem, 0, 2 * oneM, 1234ULL},
  {oneM + 2, 1, 2 * oneM, true, HistTypeGmem, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneM, false, HistTypeGmem, 0, 2 * oneM, 1234ULL},
  {oneM, 21, 2 * oneM, true, HistTypeGmem, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneM, false, HistTypeGmem, 0, 2 * oneM, 1234ULL},
  {oneM + 1, 21, 2 * oneM, true, HistTypeGmem, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneM, false, HistTypeGmem, 0, 2 * oneM, 1234ULL},
  {oneM + 2, 21, 2 * oneM, true, HistTypeGmem, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneK, false, HistTypeSmem, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmem, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmem, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmem, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmem, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmem, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmem, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmem, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmem, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmem, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmem, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmem, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneK, false, HistTypeSmemMatchAny, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmemMatchAny, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmemMatchAny, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmemMatchAny, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmemMatchAny, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmemMatchAny, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmemMatchAny, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmemMatchAny, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmemMatchAny, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmemMatchAny, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmemMatchAny, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmemMatchAny, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneK, false, HistTypeSmemBits16, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmemBits16, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmemBits16, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmemBits16, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmemBits16, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmemBits16, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmemBits16, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmemBits16, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmemBits16, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmemBits16, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmemBits16, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmemBits16, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneK, false, HistTypeSmemBits8, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmemBits8, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmemBits8, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmemBits8, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmemBits8, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmemBits8, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmemBits8, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmemBits8, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmemBits8, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmemBits8, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmemBits8, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmemBits8, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneK, false, HistTypeSmemBits4, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmemBits4, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmemBits4, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmemBits4, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmemBits4, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmemBits4, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmemBits4, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmemBits4, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmemBits4, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmemBits4, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmemBits4, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmemBits4, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneK, false, HistTypeSmemBits2, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmemBits2, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmemBits2, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmemBits2, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmemBits2, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmemBits2, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmemBits2, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmemBits2, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmemBits2, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmemBits2, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmemBits2, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmemBits2, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneK, false, HistTypeSmemBits1, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmemBits1, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmemBits1, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmemBits1, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmemBits1, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmemBits1, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmemBits1, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmemBits1, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmemBits1, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmemBits1, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmemBits1, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmemBits1, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneM, false, HistTypeSmemHash, 0, 2 * oneM, 1234ULL},
  {oneM, 1, 2 * oneM, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneM, false, HistTypeSmemHash, 0, 2 * oneM, 1234ULL},
  {oneM + 1, 1, 2 * oneM, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneM, false, HistTypeSmemHash, 0, 2 * oneM, 1234ULL},
  {oneM + 2, 1, 2 * oneM, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM, 1, 2 * oneK, false, HistTypeSmemHash, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeSmemHash, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeSmemHash, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneM, false, HistTypeSmemHash, 0, 2 * oneM, 1234ULL},
  {oneM, 21, 2 * oneM, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneM, false, HistTypeSmemHash, 0, 2 * oneM, 1234ULL},
  {oneM + 1, 21, 2 * oneM, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneM, false, HistTypeSmemHash, 0, 2 * oneM, 1234ULL},
  {oneM + 2, 21, 2 * oneM, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeSmemHash, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeSmemHash, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeSmemHash, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeSmemHash, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeSmemHash, 1000, 50, 1234ULL},

  {oneM, 1, 2 * oneM, false, HistTypeAuto, 0, 2 * oneM, 1234ULL},
  {oneM, 1, 2 * oneM, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneM, false, HistTypeAuto, 0, 2 * oneM, 1234ULL},
  {oneM + 1, 1, 2 * oneM, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneM, false, HistTypeAuto, 0, 2 * oneM, 1234ULL},
  {oneM + 2, 1, 2 * oneM, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM, 1, 2 * oneK, false, HistTypeAuto, 0, 2 * oneK, 1234ULL},
  {oneM, 1, 2 * oneK, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 1, 1, 2 * oneK, false, HistTypeAuto, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 1, 2 * oneK, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 2, 1, 2 * oneK, false, HistTypeAuto, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 1, 2 * oneK, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneM, false, HistTypeAuto, 0, 2 * oneM, 1234ULL},
  {oneM, 21, 2 * oneM, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneM, false, HistTypeAuto, 0, 2 * oneM, 1234ULL},
  {oneM + 1, 21, 2 * oneM, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneM, false, HistTypeAuto, 0, 2 * oneM, 1234ULL},
  {oneM + 2, 21, 2 * oneM, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM, 21, 2 * oneK, false, HistTypeAuto, 0, 2 * oneK, 1234ULL},
  {oneM, 21, 2 * oneK, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 1, 21, 2 * oneK, false, HistTypeAuto, 0, 2 * oneK, 1234ULL},
  {oneM + 1, 21, 2 * oneK, true, HistTypeAuto, 1000, 50, 1234ULL},
  {oneM + 2, 21, 2 * oneK, false, HistTypeAuto, 0, 2 * oneK, 1234ULL},
  {oneM + 2, 21, 2 * oneK, true, HistTypeAuto, 1000, 50, 1234ULL},
};
TEST_P(HistTest, Result) {
  ASSERT_TRUE(
    devArrMatch(ref_bins, bins, params.nbins * params.ncols, Compare<int>()));
}
INSTANTIATE_TEST_CASE_P(HistTests, HistTest, ::testing::ValuesIn(inputs));

}  // end namespace Stats
}  // end namespace MLCommon
