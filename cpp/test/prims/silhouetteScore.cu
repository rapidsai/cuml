#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>
#include <algorithm>
#include <iostream>
#include <random>
#include "common/cuml_allocator.hpp"
#include "metrics/silhouetteScore.h"
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

//parameter structure definition
struct silhouetteScoreParam {
  int nRows;
  int nCols;
  int nLabels;
  int metric;
  double tolerance;
};

//test fixture class
template <typename LabelT, typename DataT>
class silhouetteScoreTest
  : public ::testing::TestWithParam<silhouetteScoreParam> {
 protected:
  //the constructor
  void SetUp() override {
    //getting the parameters
    params = ::testing::TestWithParam<silhouetteScoreParam>::GetParam();

    nRows = params.nRows;
    nCols = params.nCols;
    nLabels = params.nLabels;
    int nElements = nRows * nCols;

    //generating random value test input
    std::vector<double> h_X(nElements, 0.0);
    std::vector<int> h_labels(nRows, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_int_distribution<int> intGenerator(0, nLabels - 1);
    std::uniform_real_distribution<double> realGenerator(0, 100);

    std::generate(h_X.begin(), h_X.end(), [&]() { return realGenerator(dre); });
    std::generate(h_labels.begin(), h_labels.end(),
                  [&]() { return intGenerator(dre); });

    //allocating and initializing memory to the GPU
    CUDA_CHECK(hipStreamCreate(&stream));
    MLCommon::allocate(d_X, nElements, true);
    MLCommon::allocate(d_labels, nElements, true);
    MLCommon::allocate(sampleSilScore, nElements);

    MLCommon::updateDevice(d_X, &h_X[0], (int)nElements, stream);
    MLCommon::updateDevice(d_labels, &h_labels[0], (int)nElements, stream);
    std::shared_ptr<MLCommon::deviceAllocator> allocator(
      new defaultDeviceAllocator);

    //finding the distance matrix

    device_buffer<double> d_distanceMatrix(allocator, stream, nRows * nRows);
    device_buffer<char> workspace(allocator, stream, 1);
    double *h_distanceMatrix =
      (double *)malloc(nRows * nRows * sizeof(double *));

    MLCommon::Distance::pairwiseDistance(
      d_X, d_X, d_distanceMatrix.data(), nRows, nRows, nCols, workspace,
      static_cast<Distance::DistanceType>(params.metric), stream);

    CUDA_CHECK(hipStreamSynchronize(stream));

    MLCommon::updateHost(h_distanceMatrix, d_distanceMatrix.data(),
                         nRows * nRows, stream);

    //finding the bincount array

    double *binCountArray = (double *)malloc(nLabels * sizeof(double *));
    memset(binCountArray, 0, nLabels * sizeof(double));

    for (int i = 0; i < nRows; ++i) {
      binCountArray[h_labels[i]] += 1;
    }

    //finding the average intra cluster distance for every element

    double *a = (double *)malloc(nRows * sizeof(double *));

    for (int i = 0; i < nRows; ++i) {
      int myLabel = h_labels[i];
      double sumOfIntraClusterD = 0;

      for (int j = 0; j < nRows; ++j) {
        if (h_labels[j] == myLabel) {
          sumOfIntraClusterD += h_distanceMatrix[i * nRows + j];
        }
      }

      if (binCountArray[myLabel] <= 1)
        a[i] = -1;
      else
        a[i] = sumOfIntraClusterD / (binCountArray[myLabel] - 1);
    }

    //finding the average inter cluster distance for every element

    double *b = (double *)malloc(nRows * sizeof(double *));

    for (int i = 0; i < nRows; ++i) {
      int myLabel = h_labels[i];
      double minAvgInterCD = ULLONG_MAX;

      for (int j = 0; j < nLabels; ++j) {
        int curClLabel = j;
        if (curClLabel == myLabel) continue;
        double avgInterCD = 0;

        for (int k = 0; k < nRows; ++k) {
          if (h_labels[k] == curClLabel) {
            avgInterCD += h_distanceMatrix[i * nRows + k];
          }
        }

        if (binCountArray[curClLabel])
          avgInterCD /= binCountArray[curClLabel];
        else
          avgInterCD = ULLONG_MAX;
        minAvgInterCD = min(minAvgInterCD, avgInterCD);
      }

      b[i] = minAvgInterCD;
    }

    //finding the silhouette score for every element

    double *truthSampleSilScore = (double *)malloc(nRows * sizeof(double *));
    for (int i = 0; i < nRows; ++i) {
      if (a[i] == -1)
        truthSampleSilScore[i] = 0;
      else if (a[i] == 0 && b[i] == 0)
        truthSampleSilScore[i] = 0;
      else
        truthSampleSilScore[i] = (b[i] - a[i]) / max(a[i], b[i]);
      truthSilhouetteScore += truthSampleSilScore[i];
    }

    truthSilhouetteScore /= nRows;

    //calling the silhouetteScore CUDA implementation
    computedSilhouetteScore = MLCommon::Metrics::silhouetteScore(
      d_X, nRows, nCols, d_labels, nLabels, sampleSilScore, allocator, stream,
      params.metric);
  }

  //the destructor
  void TearDown() override {
    CUDA_CHECK(hipFree(d_X));
    CUDA_CHECK(hipFree(d_labels));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  //declaring the data values
  silhouetteScoreParam params;
  int nLabels;
  DataT *d_X = nullptr;
  DataT *sampleSilScore = nullptr;
  LabelT *d_labels = nullptr;
  int nRows;
  int nCols;
  double truthSilhouetteScore = 0;
  double computedSilhouetteScore = 0;
  hipStream_t stream;
};

//setting test parameter values
const std::vector<silhouetteScoreParam> inputs = {
  {4, 2, 3, 0, 0.00001},  {4, 2, 2, 5, 0.00001},  {8, 8, 3, 4, 0.00001},
  {11, 2, 5, 0, 0.00001}, {40, 2, 8, 0, 0.00001}, {12, 7, 3, 2, 0.00001},
  {7, 5, 5, 3, 0.00001}};

//writing the test suite
typedef silhouetteScoreTest<int, double> silhouetteScoreTestClass;
TEST_P(silhouetteScoreTestClass, Result) {
  ASSERT_NEAR(computedSilhouetteScore, truthSilhouetteScore, params.tolerance);
}
INSTANTIATE_TEST_CASE_P(silhouetteScore, silhouetteScoreTestClass,
                        ::testing::ValuesIn(inputs));

}  //end namespace Metrics
}  //end namespace MLCommon
