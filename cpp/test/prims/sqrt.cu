#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "test_utils.h"
#include <gtest/gtest.h>
#include <linalg/sqrt.cuh>
#include <raft/cudart_utils.h>
#include <raft/random/rng.hpp>

namespace MLCommon {
namespace LinAlg {

template <typename Type>
__global__ void naiveSqrtElemKernel(Type* out, const Type* in1, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) { out[idx] = raft::mySqrt(in1[idx]); }
}

template <typename Type>
void naiveSqrtElem(Type* out, const Type* in1, int len)
{
  static const int TPB = 64;
  int nblks            = raft::ceildiv(len, TPB);
  naiveSqrtElemKernel<Type><<<nblks, TPB>>>(out, in1, len);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename T>
struct SqrtInputs {
  T tolerance;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const SqrtInputs<T>& dims)
{
  return os;
}

template <typename T>
class SqrtTest : public ::testing::TestWithParam<SqrtInputs<T>> {
 protected:
  SqrtTest() : in1(0, stream), out_ref(0, stream), out(0, stream) {}

  void SetUp() override
  {
    params = ::testing::TestWithParam<SqrtInputs<T>>::GetParam();
    raft::random::Rng r(params.seed);
    RAFT_CUDA_TRY(hipStreamCreate(&stream));
    int len = params.len;
    in1.resize(len, stream);
    out_ref.resize(len, stream);
    out.resize(len, stream);
    r.uniform(in1.data(), len, T(1.0), T(2.0), stream);

    naiveSqrtElem(out_ref.data(), in1.data(), len);

    sqrt(out.data(), in1.data(), len, stream);
    sqrt(in1.data(), in1.data(), len, stream);
    RAFT_CUDA_TRY(hipStreamDestroy(stream));
  }

 protected:
  hipStream_t stream = 0;
  SqrtInputs<T> params;
  rmm::device_uvector<T> in1, out_ref, out;
  int device_count = 0;
};

const std::vector<SqrtInputs<float>> inputsf2 = {{0.000001f, 1024 * 1024, 1234ULL}};

const std::vector<SqrtInputs<double>> inputsd2 = {{0.00000001, 1024 * 1024, 1234ULL}};

typedef SqrtTest<float> SqrtTestF;
TEST_P(SqrtTestF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(
    out_ref.data(), out.data(), params.len, raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(
    out_ref.data(), in1.data(), params.len, raft::CompareApprox<float>(params.tolerance)));
}

typedef SqrtTest<double> SqrtTestD;
TEST_P(SqrtTestD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(
    out_ref.data(), out.data(), params.len, raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(raft::devArrMatch(
    out_ref.data(), in1.data(), params.len, raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(SqrtTests, SqrtTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(SqrtTests, SqrtTestD, ::testing::ValuesIn(inputsd2));

}  // end namespace LinAlg
}  // end namespace MLCommon
