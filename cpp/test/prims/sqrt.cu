#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/sqrt.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename Type>
__global__ void naiveSqrtElemKernel(Type *out, const Type *in1, int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = mySqrt(in1[idx]);
  }
}

template <typename Type>
void naiveSqrtElem(Type *out, const Type *in1, int len) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naiveSqrtElemKernel<Type><<<nblks, TPB>>>(out, in1, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct SqrtInputs {
  T tolerance;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const SqrtInputs<T> &dims) {
  return os;
}

template <typename T>
class SqrtTest : public ::testing::TestWithParam<SqrtInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<SqrtInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    int len = params.len;
    allocate(in1, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in1, len, T(1.0), T(2.0), stream);

    naiveSqrtElem(out_ref, in1, len);

    sqrt(out, in1, len, stream);
    sqrt(in1, in1, len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

 protected:
  SqrtInputs<T> params;
  T *in1, *out_ref, *out;
  int device_count = 0;
};

const std::vector<SqrtInputs<float>> inputsf2 = {
  {0.000001f, 1024 * 1024, 1234ULL}};

const std::vector<SqrtInputs<double>> inputsd2 = {
  {0.00000001, 1024 * 1024, 1234ULL}};

typedef SqrtTest<float> SqrtTestF;
TEST_P(SqrtTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<float>(params.tolerance)));
}

typedef SqrtTest<double> SqrtTestD;
TEST_P(SqrtTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ref, in1, params.len,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(SqrtTests, SqrtTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(SqrtTests, SqrtTestD, ::testing::ValuesIn(inputsd2));

}  // end namespace LinAlg
}  // end namespace MLCommon
