/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include <functions/linearReg.cuh>
#include <raft/random/rng.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Functions {

template <typename T>
struct LinRegLossInputs {
  T tolerance;
  T n_rows;
  T n_cols;
  int len;
};

template <typename T>
class LinRegLossTest : public ::testing::TestWithParam<LinRegLossInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<LinRegLossInputs<T>>::GetParam();
    int len = params.len;
    int n_rows = params.n_rows;
    int n_cols = params.n_cols;

    T *labels, *coef;

    raft::handle_t handle;

    hipStream_t stream = handle.get_stream();

    raft::allocate(in, len);
    raft::allocate(out, 1);
    raft::allocate(out_lasso, 1);
    raft::allocate(out_ridge, 1);
    raft::allocate(out_elasticnet, 1);
    raft::allocate(out_grad, n_cols);
    raft::allocate(out_lasso_grad, n_cols);
    raft::allocate(out_ridge_grad, n_cols);
    raft::allocate(out_elasticnet_grad, n_cols);
    raft::allocate(out_ref, 1);
    raft::allocate(out_lasso_ref, 1);
    raft::allocate(out_ridge_ref, 1);
    raft::allocate(out_elasticnet_ref, 1);
    raft::allocate(out_grad_ref, n_cols);
    raft::allocate(out_lasso_grad_ref, n_cols);
    raft::allocate(out_ridge_grad_ref, n_cols);
    raft::allocate(out_elasticnet_grad_ref, n_cols);

    raft::allocate(labels, params.n_rows);
    raft::allocate(coef, params.n_cols);

    T h_in[len] = {0.1, 0.35, -0.9, -1.4, 2.0, 3.1};
    raft::update_device(in, h_in, len, stream);

    T h_labels[n_rows] = {0.3, 2.0, -1.1};
    raft::update_device(labels, h_labels, n_rows, stream);

    T h_coef[n_cols] = {0.35, -0.24};
    raft::update_device(coef, h_coef, n_cols, stream);

    T h_out_ref[1] = {1.854842};
    raft::update_device(out_ref, h_out_ref, 1, stream);

    T h_out_lasso_ref[1] = {2.2088};
    raft::update_device(out_lasso_ref, h_out_lasso_ref, 1, stream);

    T h_out_ridge_ref[1] = {1.9629};
    raft::update_device(out_ridge_ref, h_out_ridge_ref, 1, stream);

    T h_out_elasticnet_ref[1] = {2.0858};
    raft::update_device(out_elasticnet_ref, h_out_elasticnet_ref, 1, stream);

    T h_out_grad_ref[n_cols] = {-0.56995, -3.12486};
    raft::update_device(out_grad_ref, h_out_grad_ref, n_cols, stream);

    T h_out_lasso_grad_ref[n_cols] = {0.03005, -3.724866};
    raft::update_device(out_lasso_grad_ref, h_out_lasso_grad_ref, n_cols,
                        stream);

    T h_out_ridge_grad_ref[n_cols] = {-0.14995, -3.412866};
    raft::update_device(out_ridge_grad_ref, h_out_ridge_grad_ref, n_cols,
                        stream);

    T h_out_elasticnet_grad_ref[n_cols] = {-0.05995, -3.568866};
    raft::update_device(out_elasticnet_grad_ref, h_out_elasticnet_grad_ref,
                        n_cols, stream);

    T alpha = 0.6;
    T l1_ratio = 0.5;

    linearRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef, out,
                  penalty::NONE, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    linearRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                       out_grad, penalty::NONE, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    linearRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef,
                  out_lasso, penalty::L1, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    linearRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                       out_lasso_grad, penalty::L1, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    linearRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef,
                  out_ridge, penalty::L2, alpha, l1_ratio, stream);

    linearRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                       out_ridge_grad, penalty::L2, alpha, l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    linearRegLoss(handle, in, params.n_rows, params.n_cols, labels, coef,
                  out_elasticnet, penalty::ELASTICNET, alpha, l1_ratio, stream);

    linearRegLossGrads(handle, in, params.n_rows, params.n_cols, labels, coef,
                       out_elasticnet_grad, penalty::ELASTICNET, alpha,
                       l1_ratio, stream);

    raft::update_device(in, h_in, len, stream);

    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(coef));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_lasso));
    CUDA_CHECK(hipFree(out_ridge));
    CUDA_CHECK(hipFree(out_elasticnet));
    CUDA_CHECK(hipFree(out_grad));
    CUDA_CHECK(hipFree(out_lasso_grad));
    CUDA_CHECK(hipFree(out_ridge_grad));
    CUDA_CHECK(hipFree(out_elasticnet_grad));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out_lasso_ref));
    CUDA_CHECK(hipFree(out_ridge_ref));
    CUDA_CHECK(hipFree(out_elasticnet_ref));
    CUDA_CHECK(hipFree(out_grad_ref));
    CUDA_CHECK(hipFree(out_lasso_grad_ref));
    CUDA_CHECK(hipFree(out_ridge_grad_ref));
    CUDA_CHECK(hipFree(out_elasticnet_grad_ref));
  }

 protected:
  LinRegLossInputs<T> params;
  T *in;
  T *out, *out_lasso, *out_ridge, *out_elasticnet;
  T *out_ref, *out_lasso_ref, *out_ridge_ref, *out_elasticnet_ref;
  T *out_grad, *out_lasso_grad, *out_ridge_grad, *out_elasticnet_grad;
  T *out_grad_ref, *out_lasso_grad_ref, *out_ridge_grad_ref,
    *out_elasticnet_grad_ref;
  std::shared_ptr<deviceAllocator> allocator;
};

const std::vector<LinRegLossInputs<float>> inputsf = {{0.01f, 3, 2, 6}};

const std::vector<LinRegLossInputs<double>> inputsd = {{0.01, 3, 2, 6}};

typedef LinRegLossTest<float> LinRegLossTestF;
TEST_P(LinRegLossTestF, Result) {
  ASSERT_TRUE(
    devArrMatch(out_ref, out, 1, raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_grad_ref, out_grad, params.n_cols,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.n_cols,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.n_cols,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.n_cols,
                          raft::CompareApprox<float>(params.tolerance)));
}

typedef LinRegLossTest<double> LinRegLossTestD;
TEST_P(LinRegLossTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, 1,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_grad_ref, out_grad, params.n_cols,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.n_cols,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.n_cols,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.n_cols,
                          raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(LinRegLossTests, LinRegLossTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(LinRegLossTests, LinRegLossTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Functions
}  // end namespace MLCommon
