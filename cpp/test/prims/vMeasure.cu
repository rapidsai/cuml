/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <algorithm>
#include <cuml/common/cuml_allocator.hpp>
#include <iostream>
#include <metrics/vMeasure.cuh>
#include <random>
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

//parameter structure definition
struct vMeasureParam {
  int nElements;
  int lowerLabelRange;
  int upperLabelRange;
  double beta;
  bool sameArrays;
  double tolerance;
};

//test fixture class
template <typename T>
class vMeasureTest : public ::testing::TestWithParam<vMeasureParam> {
 protected:
  //the constructor
  void SetUp() override {
    //getting the parameters
    params = ::testing::TestWithParam<vMeasureParam>::GetParam();

    nElements = params.nElements;
    lowerLabelRange = params.lowerLabelRange;
    upperLabelRange = params.upperLabelRange;

    //generating random value test input
    std::vector<int> arr1(nElements, 0);
    std::vector<int> arr2(nElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_int_distribution<int> intGenerator(lowerLabelRange,
                                                    upperLabelRange);

    std::generate(arr1.begin(), arr1.end(),
                  [&]() { return intGenerator(dre); });
    if (params.sameArrays) {
      arr2 = arr1;
    } else {
      std::generate(arr2.begin(), arr2.end(),
                    [&]() { return intGenerator(dre); });
    }

    //allocating and initializing memory to the GPU

    CUDA_CHECK(hipStreamCreate(&stream));
    MLCommon::allocate(truthClusterArray, nElements, true);
    MLCommon::allocate(predClusterArray, nElements, true);

    MLCommon::updateDevice(truthClusterArray, &arr1[0], (int)nElements, stream);
    MLCommon::updateDevice(predClusterArray, &arr2[0], (int)nElements, stream);
    std::shared_ptr<MLCommon::deviceAllocator> allocator(
      new raft::mr::device::default_allocator);

    //calculating the golden output
    double truthHomogeity, truthCompleteness;

    truthHomogeity = MLCommon::Metrics::homogeneityScore(
      truthClusterArray, predClusterArray, nElements, lowerLabelRange,
      upperLabelRange, allocator, stream);
    truthCompleteness = MLCommon::Metrics::homogeneityScore(
      predClusterArray, truthClusterArray, nElements, lowerLabelRange,
      upperLabelRange, allocator, stream);

    if (truthCompleteness + truthHomogeity == 0.0)
      truthVMeasure = 0.0;
    else
      truthVMeasure = ((1 + params.beta) * truthHomogeity * truthCompleteness /
                       (params.beta * truthHomogeity + truthCompleteness));
    //calling the vMeasure CUDA implementation
    computedVMeasure = MLCommon::Metrics::vMeasure(
      truthClusterArray, predClusterArray, nElements, lowerLabelRange,
      upperLabelRange, allocator, stream, params.beta);
  }

  //the destructor
  void TearDown() override {
    CUDA_CHECK(hipFree(truthClusterArray));
    CUDA_CHECK(hipFree(predClusterArray));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  //declaring the data values
  vMeasureParam params;
  T lowerLabelRange, upperLabelRange;
  T* truthClusterArray = nullptr;
  T* predClusterArray = nullptr;
  int nElements = 0;
  double truthVMeasure = 0;
  double computedVMeasure = 0;
  hipStream_t stream;
};

//setting test parameter values
const std::vector<vMeasureParam> inputs = {
  {199, 1, 10, 1.0, false, 0.000001},  {200, 15, 100, 1.0, false, 0.000001},
  {100, 1, 20, 1.0, false, 0.000001},  {10, 1, 10, 1.0, false, 0.000001},
  {198, 1, 100, 1.0, false, 0.000001}, {300, 3, 99, 1.0, false, 0.000001},
  {199, 1, 10, 1.0, true, 0.000001},   {200, 15, 100, 1.0, true, 0.000001},
  {100, 1, 20, 1.0, true, 0.000001},   {10, 1, 10, 1.0, true, 0.000001},
  {198, 1, 100, 1.0, true, 0.000001},  {300, 3, 99, 1.0, true, 0.000001}};

//writing the test suite
typedef vMeasureTest<int> vMeasureTestClass;
TEST_P(vMeasureTestClass, Result) {
  ASSERT_NEAR(computedVMeasure, truthVMeasure, params.tolerance);
}
INSTANTIATE_TEST_CASE_P(vMeasure, vMeasureTestClass,
                        ::testing::ValuesIn(inputs));

}  //end namespace Metrics
}  //end namespace MLCommon
