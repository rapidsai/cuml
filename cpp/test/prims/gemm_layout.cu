#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "cuda_utils.h"
#include "linalg/gemm.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct GemmLayoutInputs {
  int M;
  int N;
  int K;
  bool zLayout;
  bool xLayout;
  bool yLayout;
  unsigned long long int seed;
};

// Reference GEMM implementation.
template <typename T>
__global__ void naiveGemm(T *Z, T *X, T *Y, int M, int N, int K,
                          bool isZColMajor, bool isXColMajor,
                          bool isYColMajor) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int tidy = blockIdx.y * blockDim.y + threadIdx.y;

  for (int m = tidy; m < M; m += (blockDim.y * gridDim.y)) {
    for (int n = tidx; n < N; n += (blockDim.x * gridDim.x)) {
      T temp = T(0.0);
      for (int k = 0; k < K; k++) {
        int xIndex = isXColMajor ? m + k * M : m * K + k;
        int yIndex = isYColMajor ? k + n * K : k * N + n;
        temp += X[xIndex] * Y[yIndex];
      }
      int zIndex = isZColMajor ? m + n * M : m * N + n;
      Z[zIndex] = temp;
    }
  }
}

template <typename T>
class GemmLayoutTest : public ::testing::TestWithParam<GemmLayoutInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<GemmLayoutInputs<T>>::GetParam();
    hipStream_t stream;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));
    Random::Rng r(params.seed);

    // We compute Z = X * Y and compare against reference result
    // Dimensions of X : M x K
    // Dimensions of Y : K x N
    // Dimensions of Z : M x N

    T *X = NULL;  // Argument X
    T *Y = NULL;  // Argument Y

    size_t xElems = params.M * params.K;
    size_t yElems = params.K * params.N;
    size_t zElems = params.M * params.N;

    CUDA_CHECK(hipMalloc(&X, xElems * sizeof(T)));
    CUDA_CHECK(hipMalloc(&Y, yElems * sizeof(T)));
    CUDA_CHECK(hipMalloc(&refZ, zElems * sizeof(T)));
    CUDA_CHECK(hipMalloc(&Z, zElems * sizeof(T)));

    r.uniform(X, xElems, T(-10.0), T(10.0), stream);
    r.uniform(Y, yElems, T(-10.0), T(10.0), stream);

    dim3 blocks(ceildiv<int>(params.M, 128), ceildiv<int>(params.N, 4), 1);
    dim3 threads(128, 4, 1);

    naiveGemm<<<blocks, threads>>>(refZ, X, Y, params.M, params.N, params.K,
                                   params.zLayout, params.xLayout,
                                   params.yLayout);

    gemm(handle, Z, X, Y, params.M, params.N, params.K, params.zLayout,
         params.xLayout, params.yLayout, stream);

    CUDA_CHECK(hipFree(X));
    CUDA_CHECK(hipFree(Y));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(handle));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(refZ));
    CUDA_CHECK(hipFree(Z));
  }

 protected:
  GemmLayoutInputs<T> params;
  T *refZ = NULL;  // Reference result for comparison
  T *Z = NULL;     // Computed result
};

const std::vector<GemmLayoutInputs<float>> inputsf = {
  {80, 70, 80, true, true, true, 76433ULL},
  {80, 100, 40, true, true, false, 426646ULL},
  {20, 100, 20, true, false, true, 237703ULL},
  {100, 60, 30, true, false, false, 538004ULL},
  {50, 10, 60, false, true, true, 73012ULL},
  {90, 90, 30, false, true, false, 538147ULL},
  {30, 100, 10, false, false, true, 412352ULL},
  {40, 80, 100, false, false, false, 297941ULL}};

const std::vector<GemmLayoutInputs<double>> inputsd = {
  {10, 70, 40, true, true, true, 535648ULL},
  {30, 30, 30, true, true, false, 956681ULL},
  {70, 80, 50, true, false, true, 875083ULL},
  {80, 90, 70, true, false, false, 50744ULL},
  {90, 90, 30, false, true, true, 506321ULL},
  {40, 100, 70, false, true, false, 638418ULL},
  {80, 50, 30, false, false, true, 701529ULL},
  {50, 80, 60, false, false, false, 893038ULL}};

typedef GemmLayoutTest<float> GemmLayoutTestF;
TEST_P(GemmLayoutTestF, Result) {
  ASSERT_TRUE(
    devArrMatch(refZ, Z, params.M * params.N, CompareApprox<float>(1e-4)));
}

typedef GemmLayoutTest<double> GemmLayoutTestD;
TEST_P(GemmLayoutTestD, Result) {
  ASSERT_TRUE(
    devArrMatch(refZ, Z, params.M * params.N, CompareApprox<float>(1e-6)));
}

INSTANTIATE_TEST_CASE_P(GemmLayoutTests, GemmLayoutTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(GemmLayoutTests, GemmLayoutTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace LinAlg
}  // end namespace MLCommon
