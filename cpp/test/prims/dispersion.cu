#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include "cuda_utils.h"
#include "metrics/dispersion.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

template <typename T>
struct DispersionInputs {
  T tolerance;
  int dim, clusters;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os,
                           const DispersionInputs<T> &dims) {
  return os;
}

template <typename T>
class DispersionTest : public ::testing::TestWithParam<DispersionInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<DispersionInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.clusters * params.dim;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocator.reset(new defaultDeviceAllocator);
    allocate(data, len);
    allocate(counts, params.clusters);
    allocate(exp_mean, params.dim);
    allocate(act_mean, params.dim);
    r.uniform(data, len, (T)-1.0, (T)1.0, stream);
    r.uniformInt(counts, params.clusters, 1, 100, stream);
    std::vector<int> h_counts(params.clusters, 0);
    updateHost(&(h_counts[0]), counts, params.clusters, stream);
    npoints = 0;
    for (const auto &val : h_counts) {
      npoints += val;
    }
    actualVal = dispersion(data, counts, act_mean, params.clusters, npoints,
                           params.dim, allocator, stream);
    expectedVal = T(0);
    std::vector<T> h_data(len, T(0));
    updateHost(&(h_data[0]), data, len, stream);
    std::vector<T> mean(params.dim, T(0));
    for (int i = 0; i < params.clusters; ++i) {
      for (int j = 0; j < params.dim; ++j) {
        mean[j] += h_data[i * params.dim + j] * T(h_counts[i]);
      }
    }
    for (int i = 0; i < params.dim; ++i) {
      mean[i] /= T(npoints);
    }
    updateDevice(exp_mean, &(mean[0]), params.dim, stream);
    for (int i = 0; i < params.clusters; ++i) {
      for (int j = 0; j < params.dim; ++j) {
        auto diff = h_data[i * params.dim + j] - mean[j];
        expectedVal += diff * diff * T(h_counts[i]);
      }
    }
    expectedVal = sqrt(expectedVal);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(counts));
    CUDA_CHECK(hipFree(exp_mean));
    CUDA_CHECK(hipFree(act_mean));
  }

 protected:
  DispersionInputs<T> params;
  T *data, *exp_mean, *act_mean;
  int *counts;
  hipStream_t stream;
  int npoints;
  std::shared_ptr<deviceAllocator> allocator;
  T expectedVal, actualVal;
};

const std::vector<DispersionInputs<float>> inputsf = {
  {0.001f, 10, 1000, 1234ULL},
  {0.001f, 100, 100, 1234ULL},
  {0.001f, 1000, 1000, 1234ULL}};
typedef DispersionTest<float> DispersionTestF;
TEST_P(DispersionTestF, Result) {
  auto eq = CompareApprox<float>(params.tolerance);
  ASSERT_TRUE(devArrMatch(exp_mean, act_mean, params.dim, eq));
  ASSERT_TRUE(match(expectedVal, actualVal, eq));
}
INSTANTIATE_TEST_CASE_P(DispersionTests, DispersionTestF,
                        ::testing::ValuesIn(inputsf));

const std::vector<DispersionInputs<double>> inputsd = {
  {0.001, 10, 1000, 1234ULL},
  {0.001, 100, 100, 1234ULL},
  {0.001, 1000, 1000, 1234ULL}};
typedef DispersionTest<double> DispersionTestD;
TEST_P(DispersionTestD, Result) {
  auto eq = CompareApprox<double>(params.tolerance);
  ASSERT_TRUE(devArrMatch(exp_mean, act_mean, params.dim, eq));
  ASSERT_TRUE(match(expectedVal, actualVal, eq));
}
INSTANTIATE_TEST_CASE_P(DispersionTests, DispersionTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Metrics
}  // end namespace MLCommon
