/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "add.h"
#include "linalg/add.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename InT, typename OutT = InT>
class AddTest : public ::testing::TestWithParam<AddInputs<InT, OutT>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<AddInputs<InT, OutT>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.len;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in1, len);
    allocate(in2, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in1, len, InT(-1.0), InT(1.0), stream);
    r.uniform(in2, len, InT(-1.0), InT(1.0), stream);
    naiveAddElem<InT, OutT>(out_ref, in1, in2, len);
    add<InT, OutT>(out, in1, in2, len, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                            CompareApprox<OutT>(params.tolerance)));
  }

 protected:
  AddInputs<InT, OutT> params;
  InT *in1, *in2;
  OutT *out_ref, *out;
  hipStream_t stream;
};

const std::vector<AddInputs<float>> inputsf = {
  {0.000001f, 1024 * 1024, 1234ULL},
  {0.000001f, 1024 * 1024 + 2, 1234ULL},
  {0.000001f, 1024 * 1024 + 1, 1234ULL},
};
typedef AddTest<float> AddTestF;
TEST_P(AddTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(AddTests, AddTestF, ::testing::ValuesIn(inputsf));

const std::vector<AddInputs<double>> inputsd = {
  {0.00000001, 1024 * 1024, 1234ULL},
  {0.00000001, 1024 * 1024 + 2, 1234ULL},
  {0.00000001, 1024 * 1024 + 1, 1234ULL},
};
typedef AddTest<double> AddTestD;
TEST_P(AddTestD, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(AddTests, AddTestD, ::testing::ValuesIn(inputsd));

const std::vector<AddInputs<float, double>> inputsfd = {
  {0.00000001, 1024 * 1024, 1234ULL},
  {0.00000001, 1024 * 1024 + 2, 1234ULL},
  {0.00000001, 1024 * 1024 + 1, 1234ULL},
};
typedef AddTest<float, double> AddTestFD;
TEST_P(AddTestFD, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(AddTests, AddTestFD, ::testing::ValuesIn(inputsfd));

}  // end namespace LinAlg
}  // end namespace MLCommon
