/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "cuda_utils.h"
#include "linalg/coalesced_reduction.h"
#include "random/rng.h"
#include "reduce.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct coalescedReductionInputs {
  T tolerance;
  int rows, cols;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os,
                           const coalescedReductionInputs<T> &dims) {
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T>
void coalescedReductionLaunch(T *dots, const T *data, int cols, int rows,
                              hipStream_t stream, bool inplace = false) {
  coalescedReduction(dots, data, cols, rows, (T)0, stream, inplace,
                     [] __device__(T in, int i) { return in * in; });
}

template <typename T>
class coalescedReductionTest
  : public ::testing::TestWithParam<coalescedReductionInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<coalescedReductionInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(data, len);
    allocate(dots_exp, rows);
    allocate(dots_act, rows);
    r.uniform(data, len, T(-1.0), T(1.0), stream);
    naiveCoalescedReduction(dots_exp, data, cols, rows, stream);

    // Perform reduction with default inplace = false first
    coalescedReductionLaunch(dots_act, data, cols, rows, stream);
    // Add to result with inplace = true next
    coalescedReductionLaunch(dots_act, data, cols, rows, stream, true);

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(dots_exp));
    CUDA_CHECK(hipFree(dots_act));
  }

 protected:
  coalescedReductionInputs<T> params;
  T *data, *dots_exp, *dots_act;
};

const std::vector<coalescedReductionInputs<float>> inputsf = {
  {0.000002f, 1024, 32, 1234ULL},
  {0.000002f, 1024, 64, 1234ULL},
  {0.000002f, 1024, 128, 1234ULL},
  {0.000002f, 1024, 256, 1234ULL}};

const std::vector<coalescedReductionInputs<double>> inputsd = {
  {0.000000001, 1024, 32, 1234ULL},
  {0.000000001, 1024, 64, 1234ULL},
  {0.000000001, 1024, 128, 1234ULL},
  {0.000000001, 1024, 256, 1234ULL}};

typedef coalescedReductionTest<float> coalescedReductionTestF;
TEST_P(coalescedReductionTestF, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.rows,
                          CompareApprox<float>(params.tolerance)));
}

typedef coalescedReductionTest<double> coalescedReductionTestD;
TEST_P(coalescedReductionTestD, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.rows,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(coalescedReductionTests, coalescedReductionTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(coalescedReductionTests, coalescedReductionTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace LinAlg
}  // end namespace MLCommon
