/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "matrix/math.h"
#include "random/rng.h"
#include "stats/mean.h"
#include "stats/stddev.h"
#include "test_utils.h"

namespace MLCommon {
namespace Stats {

template <typename T>
struct StdDevInputs {
  T tolerance, mean, stddev;
  int rows, cols;
  bool sample, rowMajor;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const StdDevInputs<T> &dims) {
  return os;
}

template <typename T>
class StdDevTest : public ::testing::TestWithParam<StdDevInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<StdDevInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(data, len);
    allocate(mean_act, cols);
    allocate(stddev_act, cols);
    allocate(vars_act, cols);
    r.normal(data, len, params.mean, params.stddev, stream);
    stdVarSGtest(data, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void stdVarSGtest(T *data, hipStream_t stream) {
    int rows = params.rows, cols = params.cols;

    mean(mean_act, data, cols, rows, params.sample, params.rowMajor, stream);

    stddev(stddev_act, data, mean_act, cols, rows, params.sample,
           params.rowMajor, stream);

    vars(vars_act, data, mean_act, cols, rows, params.sample, params.rowMajor,
         stream);

    Matrix::seqRoot(vars_act, T(1), cols, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(mean_act));
    CUDA_CHECK(hipFree(stddev_act));
    CUDA_CHECK(hipFree(vars_act));
  }

 protected:
  StdDevInputs<T> params;
  T *data, *mean_act, *stddev_act, *vars_act;
};

const std::vector<StdDevInputs<float>> inputsf = {
  {0.1f, 1.f, 2.f, 1024, 32, true, false, 1234ULL},
  {0.1f, 1.f, 2.f, 1024, 64, true, false, 1234ULL},
  {0.1f, 1.f, 2.f, 1024, 128, true, false, 1234ULL},
  {0.1f, 1.f, 2.f, 1024, 256, true, false, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 32, false, false, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 64, false, false, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 128, false, false, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 256, false, false, 1234ULL},
  {0.1f, 1.f, 2.f, 1024, 32, true, true, 1234ULL},
  {0.1f, 1.f, 2.f, 1024, 64, true, true, 1234ULL},
  {0.1f, 1.f, 2.f, 1024, 128, true, true, 1234ULL},
  {0.1f, 1.f, 2.f, 1024, 256, true, true, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 32, false, true, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 64, false, true, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 128, false, true, 1234ULL},
  {0.1f, -1.f, 2.f, 1024, 256, false, true, 1234ULL}};

const std::vector<StdDevInputs<double>> inputsd = {
  {0.1, 1.0, 2.0, 1024, 32, true, false, 1234ULL},
  {0.1, 1.0, 2.0, 1024, 64, true, false, 1234ULL},
  {0.1, 1.0, 2.0, 1024, 128, true, false, 1234ULL},
  {0.1, 1.0, 2.0, 1024, 256, true, false, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 32, false, false, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 64, false, false, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 128, false, false, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 256, false, false, 1234ULL},
  {0.1, 1.0, 2.0, 1024, 32, true, true, 1234ULL},
  {0.1, 1.0, 2.0, 1024, 64, true, true, 1234ULL},
  {0.1, 1.0, 2.0, 1024, 128, true, true, 1234ULL},
  {0.1, 1.0, 2.0, 1024, 256, true, true, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 32, false, true, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 64, false, true, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 128, false, true, 1234ULL},
  {0.1, -1.0, 2.0, 1024, 256, false, true, 1234ULL}};

typedef StdDevTest<float> StdDevTestF;
TEST_P(StdDevTestF, Result) {
  ASSERT_TRUE(devArrMatch(params.stddev, stddev_act, params.cols,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(stddev_act, vars_act, params.cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef StdDevTest<double> StdDevTestD;
TEST_P(StdDevTestD, Result) {
  ASSERT_TRUE(devArrMatch(params.stddev, stddev_act, params.cols,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(stddev_act, vars_act, params.cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(StdDevTests, StdDevTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(StdDevTests, StdDevTestD, ::testing::ValuesIn(inputsd));

}  // end namespace Stats
}  // end namespace MLCommon
