#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>
#include <algorithm>
#include <cuml/common/cuml_allocator.hpp>
#include <iostream>
#include <random>
#include "test_utils.h"
#include "timeSeries/jones_transform.h"

namespace MLCommon {
namespace TimeSeries {

//parameter structure definition
struct JonesTransParam {
  int batchSize;
  int pValue;
  double tolerance;
};

//test fixture class
template

  <typename DataT>
  class JonesTransTest : public ::testing::TestWithParam<JonesTransParam> {
 protected:
  //the constructor
  void SetUp() override {
    //getting the parameters
    params = ::testing::TestWithParam<JonesTransParam>::GetParam();

    nElements = params.batchSize * params.pValue;

    //generating random value test input that is stored in row major
    std::vector<double> arr1(nElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_real_distribution<double> realGenerator(0, 1);

    std::generate(arr1.begin(), arr1.end(),
                  [&]() { return realGenerator(dre); });

    //>>>>>>>>>>>>>>>>> AR transform golden output generation<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    double *newParams = (double *)malloc(nElements * sizeof(double *));
    double *tmp = (double *)malloc(params.pValue * sizeof(double *));

    //for every model in the batch
    for (int i = 0; i < params.batchSize; ++i) {
      //storing the partial autocorrelation of each ar coefficient of a given batch in newParams and the same in another temporary copy
      for (int j = 0; j < params.pValue; ++j) {
        newParams[i * params.pValue + j] =
          ((1 - exp(-1 * arr1[i * params.pValue + j])) /
           (1 + exp(-1 * arr1[i * params.pValue + j])));
        tmp[j] = newParams[i * params.pValue + j];
      }

      //calculating according to jone's recursive formula: phi(j,k) = phi(j-1,k) - a(j)*phi(j-1,j-k)
      for (int j = 1; j < params.pValue; ++j) {
        //a is partial autocorrelation for jth coefficient
        DataT a = newParams[i * params.pValue + j];

        /*the recursive implementation of the transformation with:
        - lhs tmp[k] => phi(j,k)
        - rhs tmp[k] => phi(j-1,k)
        - a => a(j)
        - newParam[i*params.pValue + j-k-1] => phi(j-1, j-k)
        */
        for (int k = 0; k < j; ++k) {
          tmp[k] -= a * newParams[i * params.pValue + (j - k - 1)];
        }

        //copying it back for the next iteration
        for (int iter = 0; iter < j; ++iter) {
          newParams[i * params.pValue + iter] = tmp[iter];
        }
      }
    }

    //allocating and initializing device memory
    CUDA_CHECK(hipStreamCreate(&stream));
    MLCommon::allocate(d_golden_ar_trans, nElements, true);
    MLCommon::allocate(d_computed_ar_trans, nElements, true);
    MLCommon::allocate(d_params, nElements, true);

    MLCommon::updateDevice(d_params, &arr1[0], (size_t)nElements, stream);
    MLCommon::updateDevice(d_golden_ar_trans, newParams, (size_t)nElements,
                           stream);
    std::shared_ptr<MLCommon::deviceAllocator> allocator(
      new defaultDeviceAllocator);

    //calling the ar_trans_param CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_params, params.batchSize,
                                          params.pValue, d_computed_ar_trans,
                                          true, false, allocator, stream);

    //>>>>>>>>>>>>>>>>> MA transform golden output generation<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    //for every model in the batch
    for (int i = 0; i < params.batchSize; ++i) {
      //storing the partial autocorrelation of each ma coefficient of a given batch in newParams and the same in another temporary copy
      for (int j = 0; j < params.pValue; ++j) {
        newParams[i * params.pValue + j] =
          ((1 - exp(-1 * arr1[i * params.pValue + j])) /
           (1 + exp(-1 * arr1[i * params.pValue + j])));
        tmp[j] = newParams[i * params.pValue + j];
      }

      //calculating according to jone's recursive formula: phi(j,k) = phi(j-1,k) - a(j)*phi(j-1,j-k)
      for (int j = 1; j < params.pValue; ++j) {
        //a is partial autocorrelation for jth coefficient
        DataT a = newParams[i * params.pValue + j];

        /*the recursive implementation of the transformation with:
        - lhs tmp[k] => phi(j,k)
        - rhs tmp[k] => phi(j-1,k)
        - a => a(j)
        - newParam[i*params.pValue + j-k-1] => phi(j-1, j-k)
        */
        for (int k = 0; k < j; ++k) {
          tmp[k] += a * newParams[i * params.pValue + (j - k - 1)];
        }

        //copying it back for the next iteration
        for (int iter = 0; iter < j; ++iter) {
          newParams[i * params.pValue + iter] = tmp[iter];
        }
      }
    }

    //allocating and initializing device memory
    MLCommon::allocate(d_golden_ma_trans, nElements, true);
    MLCommon::allocate(d_computed_ma_trans, nElements, true);

    MLCommon::updateDevice(d_golden_ma_trans, newParams, (size_t)nElements,
                           stream);

    //calling the ma_param_transform CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_params, params.batchSize,
                                          params.pValue, d_computed_ma_trans,
                                          false, false, allocator, stream);

    //>>>>>>>>>>>>>>>>> AR inverse transform <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    //allocating and initializing device memory
    MLCommon::allocate(d_computed_ar_invtrans, nElements, true);

    //calling the ar_param_inverse_transform CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_computed_ar_trans, params.batchSize,
                                          params.pValue, d_computed_ar_invtrans,
                                          true, true, allocator, stream);

    //>>>>>>>>>>>>>>>>> MA inverse transform <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    MLCommon::allocate(d_computed_ma_invtrans, nElements, true);

    //calling the ma_param_inverse_transform CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_computed_ma_trans, params.batchSize,
                                          params.pValue, d_computed_ma_invtrans,
                                          false, true, allocator, stream);
  }

  //the destructor
  void TearDown() override {
    CUDA_CHECK(hipFree(d_computed_ar_trans));
    CUDA_CHECK(hipFree(d_computed_ma_trans));
    CUDA_CHECK(hipFree(d_computed_ar_invtrans));
    CUDA_CHECK(hipFree(d_computed_ma_invtrans));
    CUDA_CHECK(hipFree(d_golden_ar_trans));
    CUDA_CHECK(hipFree(d_golden_ma_trans));
    CUDA_CHECK(hipFree(d_params));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  //declaring the data values
  JonesTransParam params;
  DataT *d_golden_ar_trans = nullptr;
  DataT *d_golden_ma_trans = nullptr;
  DataT *d_computed_ar_trans = nullptr;
  DataT *d_computed_ma_trans = nullptr;
  DataT *d_computed_ar_invtrans = nullptr;
  DataT *d_computed_ma_invtrans = nullptr;
  DataT *d_params = nullptr;
  hipStream_t stream;
  int nElements = -1;
};

//setting test parameter values
const std::vector<JonesTransParam> inputs = {
  {500, 4, 0.001},     {500, 3, 0.001},     {500, 2, 0.001},
  {500, 1, 0.001},     {5000, 4, 0.001},    {5000, 3, 0.001},
  {5000, 2, 0.001},    {5000, 1, 0.001},    {4, 4, 0.001},
  {4, 3, 0.001},       {4, 2, 0.001},       {4, 1, 0.001},
  {500000, 4, 0.0001}, {500000, 3, 0.0001}, {500000, 2, 0.0001},
  {500000, 1, 0.0001}};

//writing the test suite
typedef JonesTransTest<double> JonesTransTestClass;
TEST_P(JonesTransTestClass, Result) {
  ASSERT_TRUE(devArrMatch(d_computed_ar_trans, d_golden_ar_trans, nElements,
                          CompareApprox<double>(params.tolerance)));
  ASSERT_TRUE(devArrMatch(d_computed_ma_trans, d_golden_ma_trans, nElements,
                          CompareApprox<double>(params.tolerance)));
  /*
  Test verifying the inversion property:
  initially generated random coefficients -> ar_param_transform() / ma_param_transform() -> 
  transformed coefficients -> ar_param_inverse_transform()/ma_param_inverse_transform() -> 
  initially generated random coefficients
  */
  ASSERT_TRUE(devArrMatch(d_computed_ma_invtrans, d_params, nElements,
                          CompareApprox<double>(params.tolerance)));
  ASSERT_TRUE(devArrMatch(d_computed_ar_invtrans, d_params, nElements,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(JonesTrans, JonesTransTestClass,
                        ::testing::ValuesIn(inputs));

}  //end namespace TimeSeries
}  //end namespace MLCommon
