#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION. *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "test_utils.h"
#include <algorithm>
#include <gtest/gtest.h>
#include <iostream>
#include <raft/cudart_utils.h>
#include <raft/handle.hpp>
#include <random>
#include <rmm/device_uvector.hpp>
#include <timeSeries/jones_transform.cuh>

namespace MLCommon {
namespace TimeSeries {

// parameter structure definition
struct JonesTransParam {
  int batchSize;
  int pValue;
  double tolerance;
};

// test fixture class
template

  <typename DataT>
  class JonesTransTest : public ::testing::TestWithParam<JonesTransParam> {
 public:
  JonesTransTest()
    : params(::testing::TestWithParam<JonesTransParam>::GetParam()),
      stream(handle.get_stream()),
      nElements(params.batchSize * params.pValue),
      d_golden_ar_trans(0, stream),
      d_computed_ar_trans(0, stream),
      d_params(0, stream),
      d_golden_ma_trans(0, stream),
      d_computed_ma_trans(0, stream),
      d_computed_ar_invtrans(0, stream),
      d_computed_ma_invtrans(0, stream)
  {
  }

 protected:
  // the constructor
  void SetUp() override
  {
    // generating random value test input that is stored in row major
    std::vector<double> arr1(nElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_real_distribution<double> realGenerator(0, 1);

    std::generate(arr1.begin(), arr1.end(), [&]() { return realGenerator(dre); });

    //>>>>>>>>> AR transform golden output generation<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    double* newParams = (double*)malloc(nElements * sizeof(double*));
    double* tmp       = (double*)malloc(params.pValue * sizeof(double*));

    // for every model in the batch
    for (int i = 0; i < params.batchSize; ++i) {
      // storing the partial autocorrelation of each ar coefficient of a given batch in newParams
      // and the same in another temporary copy
      for (int j = 0; j < params.pValue; ++j) {
        newParams[i * params.pValue + j] = ((1 - exp(-1 * arr1[i * params.pValue + j])) /
                                            (1 + exp(-1 * arr1[i * params.pValue + j])));
        tmp[j]                           = newParams[i * params.pValue + j];
      }

      // calculating according to jone's recursive formula: phi(j,k) = phi(j-1,k) -
      // a(j)*phi(j-1,j-k)
      for (int j = 1; j < params.pValue; ++j) {
        // a is partial autocorrelation for jth coefficient
        DataT a = newParams[i * params.pValue + j];

        /*the recursive implementation of the transformation with:
        - lhs tmp[k] => phi(j,k)
        - rhs tmp[k] => phi(j-1,k)
        - a => a(j)
        - newParam[i*params.pValue + j-k-1] => phi(j-1, j-k)
        */
        for (int k = 0; k < j; ++k) {
          tmp[k] -= a * newParams[i * params.pValue + (j - k - 1)];
        }

        // copying it back for the next iteration
        for (int iter = 0; iter < j; ++iter) {
          newParams[i * params.pValue + iter] = tmp[iter];
        }
      }
    }

    // allocating and initializing device memory
    d_golden_ar_trans.resize(nElements, stream);
    d_computed_ar_trans.resize(nElements, stream);
    d_params.resize(nElements, stream);
    RAFT_CUDA_TRY(hipMemsetAsync(
      d_golden_ar_trans.data(), 0, d_golden_ar_trans.size() * sizeof(DataT), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(
      d_computed_ar_trans.data(), 0, d_computed_ar_trans.size() * sizeof(DataT), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(d_params.data(), 0, d_params.size() * sizeof(DataT), stream));

    raft::update_device(d_params.data(), &arr1[0], (size_t)nElements, stream);
    raft::update_device(d_golden_ar_trans.data(), newParams, (size_t)nElements, stream);

    // calling the ar_trans_param CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_params.data(),
                                          params.batchSize,
                                          params.pValue,
                                          d_computed_ar_trans.data(),
                                          true,
                                          false,
                                          stream,
                                          false);

    //>>>>>>>>> MA transform golden output generation<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    // for every model in the batch
    for (int i = 0; i < params.batchSize; ++i) {
      // storing the partial autocorrelation of each ma coefficient of a given batch in newParams
      // and the same in another temporary copy
      for (int j = 0; j < params.pValue; ++j) {
        newParams[i * params.pValue + j] = ((1 - exp(-1 * arr1[i * params.pValue + j])) /
                                            (1 + exp(-1 * arr1[i * params.pValue + j])));
        tmp[j]                           = newParams[i * params.pValue + j];
      }

      // calculating according to jone's recursive formula: phi(j,k) = phi(j-1,k) -
      // a(j)*phi(j-1,j-k)
      for (int j = 1; j < params.pValue; ++j) {
        // a is partial autocorrelation for jth coefficient
        DataT a = newParams[i * params.pValue + j];

        /*the recursive implementation of the transformation with:
        - lhs tmp[k] => phi(j,k)
        - rhs tmp[k] => phi(j-1,k)
        - a => a(j)
        - newParam[i*params.pValue + j-k-1] => phi(j-1, j-k)
        */
        for (int k = 0; k < j; ++k) {
          tmp[k] += a * newParams[i * params.pValue + (j - k - 1)];
        }

        // copying it back for the next iteration
        for (int iter = 0; iter < j; ++iter) {
          newParams[i * params.pValue + iter] = tmp[iter];
        }
      }
    }

    d_golden_ma_trans.resize(nElements, stream);
    d_computed_ma_trans.resize(nElements, stream);
    RAFT_CUDA_TRY(hipMemsetAsync(
      d_golden_ma_trans.data(), 0, d_golden_ma_trans.size() * sizeof(DataT), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(
      d_computed_ma_trans.data(), 0, d_computed_ma_trans.size() * sizeof(DataT), stream));

    raft::update_device(d_golden_ma_trans.data(), newParams, (size_t)nElements, stream);

    // calling the ma_param_transform CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_params.data(),
                                          params.batchSize,
                                          params.pValue,
                                          d_computed_ma_trans.data(),
                                          false,
                                          false,
                                          stream,
                                          false);

    //>>>>>>>>> AR inverse transform <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    d_computed_ar_invtrans.resize(nElements, stream);
    RAFT_CUDA_TRY(hipMemsetAsync(
      d_computed_ar_invtrans.data(), 0, d_computed_ar_invtrans.size() * sizeof(DataT), stream));

    // calling the ar_param_inverse_transform CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_computed_ar_trans.data(),
                                          params.batchSize,
                                          params.pValue,
                                          d_computed_ar_invtrans.data(),
                                          true,
                                          true,
                                          stream);

    //>>>>>>>>> MA inverse transform <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

    d_computed_ma_invtrans.resize(nElements, stream);
    RAFT_CUDA_TRY(hipMemsetAsync(
      d_computed_ma_invtrans.data(), 0, d_computed_ma_invtrans.size() * sizeof(DataT), stream));

    // calling the ma_param_inverse_transform CUDA implementation
    MLCommon::TimeSeries::jones_transform(d_computed_ma_trans.data(),
                                          params.batchSize,
                                          params.pValue,
                                          d_computed_ma_invtrans.data(),
                                          false,
                                          true,
                                          stream);
  }

  raft::handle_t handle;
  hipStream_t stream = 0;

  // declaring the data values
  JonesTransParam params;
  rmm::device_uvector<DataT> d_golden_ar_trans, d_golden_ma_trans, d_computed_ar_trans,
    d_computed_ma_trans, d_computed_ar_invtrans, d_computed_ma_invtrans, d_params;
  int nElements = -1;
};

// setting test parameter values
const std::vector<JonesTransParam> inputs = {{500, 4, 0.001},
                                             {500, 3, 0.001},
                                             {500, 2, 0.001},
                                             {500, 1, 0.001},
                                             {5000, 4, 0.001},
                                             {5000, 3, 0.001},
                                             {5000, 2, 0.001},
                                             {5000, 1, 0.001},
                                             {4, 4, 0.001},
                                             {4, 3, 0.001},
                                             {4, 2, 0.001},
                                             {4, 1, 0.001},
                                             {500000, 4, 0.0001},
                                             {500000, 3, 0.0001},
                                             {500000, 2, 0.0001},
                                             {500000, 1, 0.0001}};

// writing the test suite
typedef JonesTransTest<double> JonesTransTestClass;
TEST_P(JonesTransTestClass, Result)
{
  ASSERT_TRUE(raft::devArrMatch(d_golden_ar_trans.data(),
                                d_computed_ar_trans.data(),
                                nElements,
                                raft::CompareApprox<double>(params.tolerance)));
  ASSERT_TRUE(raft::devArrMatch(d_golden_ma_trans.data(),
                                d_computed_ma_trans.data(),
                                nElements,
                                raft::CompareApprox<double>(params.tolerance)));
  /*
  Test verifying the inversion property:
  initially generated random coefficients -> ar_param_transform() / ma_param_transform() ->
  transformed coefficients -> ar_param_inverse_transform()/ma_param_inverse_transform() ->
  initially generated random coefficients
  */
  ASSERT_TRUE(raft::devArrMatch(d_computed_ma_invtrans.data(),
                                d_params.data(),
                                nElements,
                                raft::CompareApprox<double>(params.tolerance)));
  ASSERT_TRUE(raft::devArrMatch(d_computed_ar_invtrans.data(),
                                d_params.data(),
                                nElements,
                                raft::CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(JonesTrans, JonesTransTestClass, ::testing::ValuesIn(inputs));

}  // end namespace TimeSeries
}  // end namespace MLCommon
