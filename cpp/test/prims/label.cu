/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "label/classlabels.h"

#include <common/cudart_utils.h>
#include <cuda_utils.h>
#include <cuml/common/cuml_allocator.hpp>
#include "test_utils.h"

#include <iostream>
#include <vector>

namespace MLCommon {
namespace Label {

class LabelTest : public ::testing::Test {
 protected:
  void SetUp() override {}
  void TearDown() override {}
};

typedef LabelTest MakeMonotonicTest;
TEST_F(MakeMonotonicTest, Result) {
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  int m = 12;

  float *data, *actual, *expected;

  allocate(data, m, true);
  allocate(actual, m, true);
  allocate(expected, m, true);

  float *data_h =
    new float[m]{1.0, 2.0, 2.0, 2.0, 2.0, 3.0, 8.0, 7.0, 8.0, 8.0, 25.0, 80.0};

  float *expected_h =
    new float[m]{1.0, 2.0, 2.0, 2.0, 2.0, 3.0, 5.0, 4.0, 5.0, 5.0, 6.0, 7.0};

  updateDevice(data, data_h, m, stream);
  updateDevice(expected, expected_h, m, stream);

  make_monotonic(actual, data, m, stream);

  CUDA_CHECK(hipStreamSynchronize(stream));

  ASSERT_TRUE(devArrMatch(actual, expected, m, Compare<bool>(), stream));

  CUDA_CHECK(hipStreamDestroy(stream));
  CUDA_CHECK(hipFree(data));
  CUDA_CHECK(hipFree(actual));

  delete data_h;
  delete expected_h;
}

TEST(LabelTest, ClassLabels) {
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  std::shared_ptr<deviceAllocator> allocator(new defaultDeviceAllocator);

  int n_rows = 6;
  float *y_d;
  allocate(y_d, n_rows);

  float y_h[] = {2, -1, 1, 2, 1, 1};
  updateDevice(y_d, y_h, n_rows, stream);

  int n_classes;
  float *y_unique_d;
  getUniqueLabels(y_d, n_rows, &y_unique_d, &n_classes, stream, allocator);

  ASSERT_EQ(n_classes, 3);

  float y_unique_exp[] = {-1, 1, 2};
  EXPECT_TRUE(devArrMatchHost(y_unique_exp, y_unique_d, n_classes,
                              Compare<float>(), stream));

  float *y_relabeled_d;
  allocate(y_relabeled_d, n_rows);

  getOvrLabels(y_d, n_rows, y_unique_d, n_classes, y_relabeled_d, 2, stream);

  float y_relabeled_exp[] = {1, -1, -1, 1, -1, -1};
  EXPECT_TRUE(devArrMatchHost(y_relabeled_exp, y_relabeled_d, n_rows,
                              Compare<float>(), stream));

  CUDA_CHECK(hipStreamDestroy(stream));
  CUDA_CHECK(hipFree(y_d));
  CUDA_CHECK(hipFree(y_unique_d));
  CUDA_CHECK(hipFree(y_relabeled_d));
}
};  // namespace Label
};  // namespace MLCommon
