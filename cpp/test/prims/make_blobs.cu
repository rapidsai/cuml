#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "test_utils.h"
#include <hipcub/hipcub.hpp>
#include <gtest/gtest.h>
#include <raft/cuda_utils.cuh>
#include <raft/cudart_utils.h>
#include <random/make_blobs.cuh>

namespace MLCommon {
namespace Random {

template <typename T>
__global__ void meanKernel(T* out,
                           int* lens,
                           const T* data,
                           const int* labels,
                           int nrows,
                           int ncols,
                           int nclusters,
                           bool row_major)
{
  int tid   = threadIdx.x + blockIdx.x * blockDim.x;
  int rowid = row_major ? tid / ncols : tid % nrows;
  int colid = row_major ? tid % ncols : tid / nrows;
  if (rowid < nrows && colid < ncols) {
    T val     = data[tid];
    int label = labels[rowid];
    int idx   = row_major ? label * ncols + colid : colid * nclusters + label;
    raft::myAtomicAdd(out + idx * 2, val);
    raft::myAtomicAdd(out + idx * 2 + 1, val * val);
    if (colid == 0) { raft::myAtomicAdd(lens + label, 1); }
  }
}

template <typename T>
__global__ void compute_mean_var(
  T* out, const T* stats, int* lens, int nrows, int ncols, bool row_major)
{
  int tid    = threadIdx.x + blockIdx.x * blockDim.x;
  int rowid  = row_major ? tid / ncols : tid % nrows;
  int colid  = row_major ? tid % ncols : tid / nrows;
  int stride = nrows * ncols;
  if (rowid < nrows && colid < ncols) {
    int len           = lens[rowid];
    auto mean         = stats[tid * 2] / len;
    out[tid]          = mean;
    out[tid + stride] = (stats[tid * 2 + 1] / len) - (mean * mean);
  }
}

template <typename T>
struct MakeBlobsInputs {
  T tolerance;
  int rows, cols, n_clusters;
  T std;
  bool row_major, shuffle;
  raft::random::GeneratorType gtype;
  uint64_t seed;
};

template <typename T>
class MakeBlobsTest : public ::testing::TestWithParam<MakeBlobsInputs<T>> {
 public:
  MakeBlobsTest()
    : params(::testing::TestWithParam<MakeBlobsInputs<T>>::GetParam()),
      stream(handle.get_stream()),
      mu_vec(params.cols * params.n_clusters, stream),
      mean_var(2 * params.n_clusters * params.cols, stream)
  {
  }

 protected:
  void SetUp() override
  {
    // Tests are configured with their expected test-values sigma. For example,
    // 4 x sigma indicates the test shouldn't fail 99.9% of the time.
    num_sigma = 50;
    auto len  = params.rows * params.cols;
    raft::random::Rng r(params.seed, params.gtype);

    rmm::device_uvector<T> data(len, stream);
    rmm::device_uvector<int> labels(params.rows, stream);
    rmm::device_uvector<T> stats(2 * params.n_clusters * params.cols, stream);
    rmm::device_uvector<int> lens(params.n_clusters, stream);

    RAFT_CUDA_TRY(hipMemsetAsync(stats.data(), 0, stats.size() * sizeof(T), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(mean_var.data(), 0, mean_var.size() * sizeof(T), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(lens.data(), 0, lens.size() * sizeof(int), stream));

    r.uniform(mu_vec.data(), params.cols * params.n_clusters, T(-10.0), T(10.0), stream);
    T* sigma_vec = nullptr;
    make_blobs(data.data(),
               labels.data(),
               params.rows,
               params.cols,
               params.n_clusters,
               stream,
               params.row_major,
               mu_vec.data(),
               sigma_vec,
               params.std,
               params.shuffle,
               T(-10.0),
               T(10.0),
               params.seed,
               params.gtype);
    static const int threads = 128;
    meanKernel<T><<<raft::ceildiv(len, threads), threads, 0, stream>>>(stats.data(),
                                                                       lens.data(),
                                                                       data.data(),
                                                                       labels.data(),
                                                                       params.rows,
                                                                       params.cols,
                                                                       params.n_clusters,
                                                                       params.row_major);
    int len1 = params.n_clusters * params.cols;
    compute_mean_var<T><<<raft::ceildiv(len1, threads), threads, 0, stream>>>(
      mean_var.data(), stats.data(), lens.data(), params.n_clusters, params.cols, params.row_major);
  }

  void check()
  {
    int len      = params.n_clusters * params.cols;
    auto compare = raft::CompareApprox<T>(num_sigma * params.tolerance);
    ASSERT_TRUE(raft::devArrMatch(mu_vec.data(), mean_var.data(), len, compare));
    ASSERT_TRUE(raft::devArrMatch(params.std, mean_var.data() + len, len, compare));
  }

 protected:
  MakeBlobsInputs<T> params;
  raft::handle_t handle;
  hipStream_t stream = 0;

  rmm::device_uvector<T> mu_vec, mean_var;
  int num_sigma;
};

typedef MakeBlobsTest<float> MakeBlobsTestF;
const std::vector<MakeBlobsInputs<float>> inputsf_t = {
  {0.0055, 1024, 32, 3, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, raft::random::GenKiss99, 1234ULL},

  {0.0055, 5003, 32, 5, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, raft::random::GenKiss99, 1234ULL},
};

TEST_P(MakeBlobsTestF, Result) { check(); }
INSTANTIATE_TEST_CASE_P(MakeBlobsTests, MakeBlobsTestF, ::testing::ValuesIn(inputsf_t));

typedef MakeBlobsTest<double> MakeBlobsTestD;
const std::vector<MakeBlobsInputs<double>> inputsd_t = {
  {0.0055, 1024, 32, 3, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, raft::random::GenKiss99, 1234ULL},

  {0.0055, 5003, 32, 5, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, raft::random::GenKiss99, 1234ULL},
};
TEST_P(MakeBlobsTestD, Result) { check(); }
INSTANTIATE_TEST_CASE_P(MakeBlobsTests, MakeBlobsTestD, ::testing::ValuesIn(inputsd_t));

}  // end namespace Random
}  // end namespace MLCommon
