#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <hipcub/hipcub.hpp>
#include <cuda_utils.cuh>
#include <random/make_blobs.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Random {

template <typename T>
__global__ void meanKernel(T* out, int* lens, const T* data, const int* labels,
                           int nrows, int ncols, int nclusters,
                           bool row_major) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int rowid = row_major ? tid / ncols : tid % nrows;
  int colid = row_major ? tid % ncols : tid / nrows;
  if (rowid < nrows && colid < ncols) {
    T val = data[tid];
    int label = labels[rowid];
    int idx = row_major ? label * ncols + colid : colid * nclusters + label;
    myAtomicAdd(out + idx * 2, val);
    myAtomicAdd(out + idx * 2 + 1, val * val);
    if (colid == 0) {
      myAtomicAdd(lens + label, 1);
    }
  }
}

template <typename T>
__global__ void compute_mean_var(T* out, const T* stats, int* lens, int nrows,
                                 int ncols, bool row_major) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int rowid = row_major ? tid / ncols : tid % nrows;
  int colid = row_major ? tid % ncols : tid / nrows;
  int stride = nrows * ncols;
  if (rowid < nrows && colid < ncols) {
    int len = lens[rowid];
    auto mean = stats[tid * 2] / len;
    out[tid] = mean;
    out[tid + stride] = (stats[tid * 2 + 1] / len) - (mean * mean);
  }
}

template <typename T>
struct MakeBlobsInputs {
  T tolerance;
  int rows, cols, n_clusters;
  T std;
  bool row_major, shuffle;
  GeneratorType gtype;
  uint64_t seed;
};

template <typename T>
class MakeBlobsTest : public ::testing::TestWithParam<MakeBlobsInputs<T>> {
 protected:
  void SetUp() override {
    // Tests are configured with their expected test-values sigma. For example,
    // 4 x sigma indicates the test shouldn't fail 99.9% of the time.
    num_sigma = 50;
    allocator.reset(new raft::mr::device::default_allocator);
    params = ::testing::TestWithParam<MakeBlobsInputs<T>>::GetParam();
    int len = params.rows * params.cols;
    CUDA_CHECK(hipStreamCreate(&stream));
    Rng r(params.seed, params.gtype);
    allocate(data, len);
    allocate(labels, params.rows);
    allocate(stats, 2 * params.n_clusters * params.cols, true);
    allocate(mean_var, 2 * params.n_clusters * params.cols, true);
    allocate(mu_vec, params.cols * params.n_clusters);
    allocate(lens, params.n_clusters, true);
    r.uniform(mu_vec, params.cols * params.n_clusters, T(-10.0), T(10.0),
              stream);
    T* sigma_vec = nullptr;
    make_blobs(data, labels, params.rows, params.cols, params.n_clusters,
               allocator, stream, params.row_major, mu_vec, sigma_vec,
               params.std, params.shuffle, T(-10.0), T(10.0), params.seed,
               params.gtype);
    static const int threads = 128;
    meanKernel<T><<<ceildiv(len, threads), threads, 0, stream>>>(
      stats, lens, data, labels, params.rows, params.cols, params.n_clusters,
      params.row_major);
    int len1 = params.n_clusters * params.cols;
    compute_mean_var<T><<<ceildiv(len1, threads), threads, 0, stream>>>(
      mean_var, stats, lens, params.n_clusters, params.cols, params.row_major);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(stats));
    CUDA_CHECK(hipFree(mu_vec));
  }

  void check() {
    int len = params.n_clusters * params.cols;
    auto compare = CompareApprox<T>(num_sigma * params.tolerance);
    ASSERT_TRUE(devArrMatch(mu_vec, mean_var, len, compare));
    ASSERT_TRUE(devArrMatch(params.std, mean_var + len, len, compare));
  }

 protected:
  hipStream_t stream;
  MakeBlobsInputs<T> params;
  int *labels, *lens;
  T *data, *stats, *mu_vec, *mean_var;
  std::shared_ptr<deviceAllocator> allocator;
  int num_sigma;
};

typedef MakeBlobsTest<float> MakeBlobsTestF;
const std::vector<MakeBlobsInputs<float>> inputsf_t = {
  {0.0055, 1024, 32, 3, 1.f, true, false, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, false, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, false, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, GenKiss99, 1234ULL},

  {0.0055, 5003, 32, 5, 1.f, true, false, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, false, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, false, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, GenKiss99, 1234ULL},
};

TEST_P(MakeBlobsTestF, Result) { check(); }
INSTANTIATE_TEST_CASE_P(MakeBlobsTests, MakeBlobsTestF,
                        ::testing::ValuesIn(inputsf_t));

typedef MakeBlobsTest<double> MakeBlobsTestD;
const std::vector<MakeBlobsInputs<double>> inputsd_t = {
  {0.0055, 1024, 32, 3, 1.0, true, false, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, false, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, false, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, GenKiss99, 1234ULL},

  {0.0055, 5003, 32, 5, 1.0, true, false, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, false, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, false, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, GenKiss99, 1234ULL},
};
TEST_P(MakeBlobsTestD, Result) { check(); }
INSTANTIATE_TEST_CASE_P(MakeBlobsTests, MakeBlobsTestD,
                        ::testing::ValuesIn(inputsd_t));

}  // end namespace Random
}  // end namespace MLCommon
