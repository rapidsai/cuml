#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <hipcub/hipcub.hpp>
#include <raft/cuda_utils.cuh>
#include <random/make_blobs.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Random {

template <typename T>
__global__ void meanKernel(T* out, int* lens, const T* data, const int* labels,
                           int nrows, int ncols, int nclusters,
                           bool row_major) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int rowid = row_major ? tid / ncols : tid % nrows;
  int colid = row_major ? tid % ncols : tid / nrows;
  if (rowid < nrows && colid < ncols) {
    T val = data[tid];
    int label = labels[rowid];
    int idx = row_major ? label * ncols + colid : colid * nclusters + label;
    raft::myAtomicAdd(out + idx * 2, val);
    raft::myAtomicAdd(out + idx * 2 + 1, val * val);
    if (colid == 0) {
      raft::myAtomicAdd(lens + label, 1);
    }
  }
}

template <typename T>
__global__ void compute_mean_var(T* out, const T* stats, int* lens, int nrows,
                                 int ncols, bool row_major) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int rowid = row_major ? tid / ncols : tid % nrows;
  int colid = row_major ? tid % ncols : tid / nrows;
  int stride = nrows * ncols;
  if (rowid < nrows && colid < ncols) {
    int len = lens[rowid];
    auto mean = stats[tid * 2] / len;
    out[tid] = mean;
    out[tid + stride] = (stats[tid * 2 + 1] / len) - (mean * mean);
  }
}

template <typename T>
struct MakeBlobsInputs {
  T tolerance;
  int rows, cols, n_clusters;
  T std;
  bool row_major, shuffle;
  raft::random::GeneratorType gtype;
  uint64_t seed;
};

template <typename T>
class MakeBlobsTest : public ::testing::TestWithParam<MakeBlobsInputs<T>> {
 protected:
  void SetUp() override {
    // Tests are configured with their expected test-values sigma. For example,
    // 4 x sigma indicates the test shouldn't fail 99.9% of the time.
    num_sigma = 50;
    allocator.reset(new raft::mr::device::default_allocator);
    params = ::testing::TestWithParam<MakeBlobsInputs<T>>::GetParam();
    int len = params.rows * params.cols;
    CUDA_CHECK(hipStreamCreate(&stream));
    raft::random::Rng r(params.seed, params.gtype);
    raft::allocate(data, len);
    raft::allocate(labels, params.rows);
    raft::allocate(stats, 2 * params.n_clusters * params.cols, true);
    raft::allocate(mean_var, 2 * params.n_clusters * params.cols, true);
    raft::allocate(mu_vec, params.cols * params.n_clusters);
    raft::allocate(lens, params.n_clusters, true);
    r.uniform(mu_vec, params.cols * params.n_clusters, T(-10.0), T(10.0),
              stream);
    T* sigma_vec = nullptr;
    make_blobs(data, labels, params.rows, params.cols, params.n_clusters,
               allocator, stream, params.row_major, mu_vec, sigma_vec,
               params.std, params.shuffle, T(-10.0), T(10.0), params.seed,
               params.gtype);
    static const int threads = 128;
    meanKernel<T><<<raft::ceildiv(len, threads), threads, 0, stream>>>(
      stats, lens, data, labels, params.rows, params.cols, params.n_clusters,
      params.row_major);
    int len1 = params.n_clusters * params.cols;
    compute_mean_var<T><<<raft::ceildiv(len1, threads), threads, 0, stream>>>(
      mean_var, stats, lens, params.n_clusters, params.cols, params.row_major);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(stats));
    CUDA_CHECK(hipFree(mu_vec));
  }

  void check() {
    int len = params.n_clusters * params.cols;
    auto compare = raft::CompareApprox<T>(num_sigma * params.tolerance);
    ASSERT_TRUE(raft::devArrMatch(mu_vec, mean_var, len, compare));
    ASSERT_TRUE(raft::devArrMatch(params.std, mean_var + len, len, compare));
  }

 protected:
  hipStream_t stream;
  MakeBlobsInputs<T> params;
  int *labels, *lens;
  T *data, *stats, *mu_vec, *mean_var;
  std::shared_ptr<deviceAllocator> allocator;
  int num_sigma;
};

typedef MakeBlobsTest<float> MakeBlobsTestF;
const std::vector<MakeBlobsInputs<float>> inputsf_t = {
  {0.0055, 1024, 32, 3, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.f, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.f, false, true, raft::random::GenKiss99, 1234ULL},

  {0.0055, 5003, 32, 5, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.f, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.f, false, true, raft::random::GenKiss99, 1234ULL},
};

TEST_P(MakeBlobsTestF, Result) { check(); }
INSTANTIATE_TEST_CASE_P(MakeBlobsTests, MakeBlobsTestF,
                        ::testing::ValuesIn(inputsf_t));

typedef MakeBlobsTest<double> MakeBlobsTestD;
const std::vector<MakeBlobsInputs<double>> inputsd_t = {
  {0.0055, 1024, 32, 3, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 1024, 32, 3, 1.0, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 1024, 8, 3, 1.0, false, true, raft::random::GenKiss99, 1234ULL},

  {0.0055, 5003, 32, 5, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, false, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, true, true, raft::random::GenKiss99, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, raft::random::GenPhilox, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, raft::random::GenTaps, 1234ULL},
  {0.0055, 5003, 32, 5, 1.0, false, true, raft::random::GenKiss99, 1234ULL},
  {0.011, 5003, 8, 5, 1.0, false, true, raft::random::GenKiss99, 1234ULL},
};
TEST_P(MakeBlobsTestD, Result) { check(); }
INSTANTIATE_TEST_CASE_P(MakeBlobsTests, MakeBlobsTestD,
                        ::testing::ValuesIn(inputsd_t));

}  // end namespace Random
}  // end namespace MLCommon
