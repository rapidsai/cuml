/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <functions/penalty.cuh>
#include <raft/random/rng.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Functions {

template <typename T>
struct PenaltyInputs {
  T tolerance;
  int len;
};

template <typename T>
class PenaltyTest : public ::testing::TestWithParam<PenaltyInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<PenaltyInputs<T>>::GetParam();
    int len = params.len;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    raft::allocate(in, len);
    raft::allocate(out_lasso, 1);
    raft::allocate(out_ridge, 1);
    raft::allocate(out_elasticnet, 1);
    raft::allocate(out_lasso_grad, len);
    raft::allocate(out_ridge_grad, len);
    raft::allocate(out_elasticnet_grad, len);
    raft::allocate(out_lasso_ref, 1);
    raft::allocate(out_ridge_ref, 1);
    raft::allocate(out_elasticnet_ref, 1);
    raft::allocate(out_lasso_grad_ref, len);
    raft::allocate(out_ridge_grad_ref, len);
    raft::allocate(out_elasticnet_grad_ref, len);

    T h_in[len] = {0.1, 0.35, -0.9, -1.4};
    raft::update_device(in, h_in, len, stream);

    T h_out_lasso_ref[1] = {1.65};
    raft::update_device(out_lasso_ref, h_out_lasso_ref, 1, stream);

    T h_out_ridge_ref[1] = {1.741499};
    raft::update_device(out_ridge_ref, h_out_ridge_ref, 1, stream);

    T h_out_elasticnet_ref[1] = {1.695749};
    raft::update_device(out_elasticnet_ref, h_out_elasticnet_ref, 1, stream);

    T h_out_lasso_grad_ref[len] = {0.6, 0.6, -0.6, -0.6};
    raft::update_device(out_lasso_grad_ref, h_out_lasso_grad_ref, len, stream);

    T h_out_ridge_grad_ref[len] = {0.12, 0.42, -1.08, -1.68};
    raft::update_device(out_ridge_grad_ref, h_out_ridge_grad_ref, len, stream);

    T h_out_elasticnet_grad_ref[len] = {0.36, 0.51, -0.84, -1.14};
    raft::update_device(out_elasticnet_grad_ref, h_out_elasticnet_grad_ref, len,
                        stream);

    T alpha = 0.6;
    T l1_ratio = 0.5;

    lasso(out_lasso, in, len, alpha, stream);
    ridge(out_ridge, in, len, alpha, stream);
    elasticnet(out_elasticnet, in, len, alpha, l1_ratio, stream);
    lassoGrad(out_lasso_grad, in, len, alpha, stream);
    ridgeGrad(out_ridge_grad, in, len, alpha, stream);
    elasticnetGrad(out_elasticnet_grad, in, len, alpha, l1_ratio, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_lasso));
    CUDA_CHECK(hipFree(out_ridge));
    CUDA_CHECK(hipFree(out_elasticnet));
    CUDA_CHECK(hipFree(out_lasso_grad));
    CUDA_CHECK(hipFree(out_ridge_grad));
    CUDA_CHECK(hipFree(out_elasticnet_grad));
    CUDA_CHECK(hipFree(out_lasso_ref));
    CUDA_CHECK(hipFree(out_ridge_ref));
    CUDA_CHECK(hipFree(out_elasticnet_ref));
    CUDA_CHECK(hipFree(out_lasso_grad_ref));
    CUDA_CHECK(hipFree(out_ridge_grad_ref));
    CUDA_CHECK(hipFree(out_elasticnet_grad_ref));
  }

 protected:
  PenaltyInputs<T> params;
  T *in;
  T *out_lasso, *out_ridge, *out_elasticnet;
  T *out_lasso_ref, *out_ridge_ref, *out_elasticnet_ref;
  T *out_lasso_grad, *out_ridge_grad, *out_elasticnet_grad;
  T *out_lasso_grad_ref, *out_ridge_grad_ref, *out_elasticnet_grad_ref;
};

const std::vector<PenaltyInputs<float>> inputsf = {{0.01f, 4}};

const std::vector<PenaltyInputs<double>> inputsd = {{0.01, 4}};

typedef PenaltyTest<float> PenaltyTestF;
TEST_P(PenaltyTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.len,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.len,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          raft::CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.len,
                          raft::CompareApprox<float>(params.tolerance)));
}

typedef PenaltyTest<double> PenaltyTestD;
TEST_P(PenaltyTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.len,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.len,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          raft::CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.len,
                          raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(PenaltyTests, PenaltyTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(PenaltyTests, PenaltyTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Functions
}  // end namespace MLCommon
