#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "functions/penalty.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace Functions {

template <typename T>
struct PenaltyInputs {
  T tolerance;
  int len;
};

template <typename T>
class PenaltyTest : public ::testing::TestWithParam<PenaltyInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<PenaltyInputs<T>>::GetParam();
    int len = params.len;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    allocate(in, len);
    allocate(out_lasso, 1);
    allocate(out_ridge, 1);
    allocate(out_elasticnet, 1);
    allocate(out_lasso_grad, len);
    allocate(out_ridge_grad, len);
    allocate(out_elasticnet_grad, len);
    allocate(out_lasso_ref, 1);
    allocate(out_ridge_ref, 1);
    allocate(out_elasticnet_ref, 1);
    allocate(out_lasso_grad_ref, len);
    allocate(out_ridge_grad_ref, len);
    allocate(out_elasticnet_grad_ref, len);

    T h_in[len] = {0.1, 0.35, -0.9, -1.4};
    updateDevice(in, h_in, len, stream);

    T h_out_lasso_ref[1] = {1.65};
    updateDevice(out_lasso_ref, h_out_lasso_ref, 1, stream);

    T h_out_ridge_ref[1] = {1.741499};
    updateDevice(out_ridge_ref, h_out_ridge_ref, 1, stream);

    T h_out_elasticnet_ref[1] = {1.695749};
    updateDevice(out_elasticnet_ref, h_out_elasticnet_ref, 1, stream);

    T h_out_lasso_grad_ref[len] = {0.6, 0.6, -0.6, -0.6};
    updateDevice(out_lasso_grad_ref, h_out_lasso_grad_ref, len, stream);

    T h_out_ridge_grad_ref[len] = {0.12, 0.42, -1.08, -1.68};
    updateDevice(out_ridge_grad_ref, h_out_ridge_grad_ref, len, stream);

    T h_out_elasticnet_grad_ref[len] = {0.36, 0.51, -0.84, -1.14};
    updateDevice(out_elasticnet_grad_ref, h_out_elasticnet_grad_ref, len,
                 stream);

    T alpha = 0.6;
    T l1_ratio = 0.5;

    lasso(out_lasso, in, len, alpha, stream);
    ridge(out_ridge, in, len, alpha, stream);
    elasticnet(out_elasticnet, in, len, alpha, l1_ratio, stream);
    lassoGrad(out_lasso_grad, in, len, alpha, stream);
    ridgeGrad(out_ridge_grad, in, len, alpha, stream);
    elasticnetGrad(out_elasticnet_grad, in, len, alpha, l1_ratio, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_lasso));
    CUDA_CHECK(hipFree(out_ridge));
    CUDA_CHECK(hipFree(out_elasticnet));
    CUDA_CHECK(hipFree(out_lasso_grad));
    CUDA_CHECK(hipFree(out_ridge_grad));
    CUDA_CHECK(hipFree(out_elasticnet_grad));
    CUDA_CHECK(hipFree(out_lasso_ref));
    CUDA_CHECK(hipFree(out_ridge_ref));
    CUDA_CHECK(hipFree(out_elasticnet_ref));
    CUDA_CHECK(hipFree(out_lasso_grad_ref));
    CUDA_CHECK(hipFree(out_ridge_grad_ref));
    CUDA_CHECK(hipFree(out_elasticnet_grad_ref));
  }

 protected:
  PenaltyInputs<T> params;
  T *in;
  T *out_lasso, *out_ridge, *out_elasticnet;
  T *out_lasso_ref, *out_ridge_ref, *out_elasticnet_ref;
  T *out_lasso_grad, *out_ridge_grad, *out_elasticnet_grad;
  T *out_lasso_grad_ref, *out_ridge_grad_ref, *out_elasticnet_grad_ref;
};

const std::vector<PenaltyInputs<float>> inputsf = {{0.01f, 4}};

const std::vector<PenaltyInputs<double>> inputsd = {{0.01, 4}};

typedef PenaltyTest<float> PenaltyTestF;
TEST_P(PenaltyTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.len,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.len,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          CompareApprox<float>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.len, CompareApprox<float>(params.tolerance)));
}

typedef PenaltyTest<double> PenaltyTestD;
TEST_P(PenaltyTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_lasso_ref, out_lasso, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_lasso_grad_ref, out_lasso_grad, params.len,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_ref, out_ridge, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_ridge_grad_ref, out_ridge_grad, params.len,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_ref, out_elasticnet, 1,
                          CompareApprox<double>(params.tolerance)));

  ASSERT_TRUE(devArrMatch(out_elasticnet_grad_ref, out_elasticnet_grad,
                          params.len, CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(PenaltyTests, PenaltyTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(PenaltyTests, PenaltyTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace Functions
}  // end namespace MLCommon
