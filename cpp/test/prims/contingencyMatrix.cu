#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "test_utils.h"
#include <algorithm>
#include <gtest/gtest.h>
#include <iostream>
#include <metrics/contingencyMatrix.cuh>
#include <raft/cudart_utils.h>
#include <random>
#include <rmm/device_uvector.hpp>

namespace MLCommon {
namespace Metrics {

struct ContingencyMatrixParam {
  int nElements;
  int minClass;
  int maxClass;
  bool calcCardinality;
  bool skipLabels;
  float tolerance;
};

template <typename T>
class ContingencyMatrixTest : public ::testing::TestWithParam<ContingencyMatrixParam> {
 protected:
  ContingencyMatrixTest()
    : pWorkspace(0, stream),
      dY(0, stream),
      dYHat(0, stream),
      dComputedOutput(0, stream),
      dGoldenOutput(0, stream)
  {
  }

  void SetUp() override
  {
    params = ::testing::TestWithParam<ContingencyMatrixParam>::GetParam();

    int numElements     = params.nElements;
    int lowerLabelRange = params.minClass;
    int upperLabelRange = params.maxClass;

    std::vector<int> y(numElements, 0);
    std::vector<int> y_hat(numElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_int_distribution<int> intGenerator(lowerLabelRange, upperLabelRange);

    std::generate(y.begin(), y.end(), [&]() { return intGenerator(dre); });
    std::generate(y_hat.begin(), y_hat.end(), [&]() { return intGenerator(dre); });

    if (params.skipLabels) {
      // remove two label value from input arrays
      int y1 = (upperLabelRange - lowerLabelRange) / 2;
      int y2 = y1 + (upperLabelRange - lowerLabelRange) / 4;

      // replacement values
      int y1_R = y1 + 1;
      int y2_R = y2 + 1;

      std::replace(y.begin(), y.end(), y1, y1_R);
      std::replace(y.begin(), y.end(), y2, y2_R);
      std::replace(y_hat.begin(), y_hat.end(), y1, y1_R);
      std::replace(y_hat.begin(), y_hat.end(), y2, y2_R);
    }

    RAFT_CUDA_TRY(hipStreamCreate(&stream));
    dY.resize(numElements, stream);
    dYHat.resize(numElements, stream);

    raft::update_device(dYHat.data(), &y_hat[0], numElements, stream);
    raft::update_device(dY.data(), &y[0], numElements, stream);

    if (params.calcCardinality) {
      MLCommon::Metrics::getInputClassCardinality(
        dY.data(), numElements, stream, minLabel, maxLabel);
    } else {
      minLabel = lowerLabelRange;
      maxLabel = upperLabelRange;
    }

    numUniqueClasses = maxLabel - minLabel + 1;

    dComputedOutput.resize(numUniqueClasses * numUniqueClasses, stream);
    dGoldenOutput.resize(numUniqueClasses * numUniqueClasses, stream);

    // generate golden output on CPU
    size_t sizeOfMat = numUniqueClasses * numUniqueClasses * sizeof(int);
    std::vector<int> hGoldenOutput(sizeOfMat, 0);

    for (int i = 0; i < numElements; i++) {
      auto row    = y[i] - minLabel;
      auto column = y_hat[i] - minLabel;
      hGoldenOutput[row * numUniqueClasses + column] += 1;
    }

    raft::update_device(
      dGoldenOutput.data(), hGoldenOutput.data(), numUniqueClasses * numUniqueClasses, stream);

    workspaceSz = MLCommon::Metrics::getContingencyMatrixWorkspaceSize(
      numElements, dY.data(), stream, minLabel, maxLabel);
    pWorkspace.resize(workspaceSz, stream);
    raft::interruptible::synchronize(stream);
  }

  void TearDown() override { RAFT_CUDA_TRY(hipStreamDestroy(stream)); }

  void RunTest()
  {
    int numElements = params.nElements;
    MLCommon::Metrics::contingencyMatrix(dY.data(),
                                         dYHat.data(),
                                         numElements,
                                         dComputedOutput.data(),
                                         stream,
                                         (void*)pWorkspace.data(),
                                         workspaceSz,
                                         minLabel,
                                         maxLabel);
    ASSERT_TRUE(raft::devArrMatch(dComputedOutput.data(),
                                  dGoldenOutput.data(),
                                  numUniqueClasses * numUniqueClasses,
                                  raft::Compare<T>()));
  }

  ContingencyMatrixParam params;
  int numUniqueClasses = -1;
  T minLabel, maxLabel;
  hipStream_t stream = 0;
  size_t workspaceSz;
  rmm::device_uvector<char> pWorkspace;
  rmm::device_uvector<T> dY, dYHat;
  rmm::device_uvector<int> dComputedOutput, dGoldenOutput;
};

const std::vector<ContingencyMatrixParam> inputs = {
  {10000, 1, 10, true, false, 0.000001},
  {10000, 1, 5000, true, false, 0.000001},
  {10000, 1, 10000, true, false, 0.000001},
  {10000, 1, 20000, true, false, 0.000001},
  {10000, 1, 10, false, false, 0.000001},
  {10000, 1, 5000, false, false, 0.000001},
  {10000, 1, 10000, false, false, 0.000001},
  {10000, 1, 20000, false, false, 0.000001},
  {100000, 1, 100, false, false, 0.000001},
  {1000000, 1, 1200, true, false, 0.000001},
  {1000000, 1, 10000, false, false, 0.000001},
  {100000, 1, 100, false, true, 0.000001},
};

typedef ContingencyMatrixTest<int> ContingencyMatrixTestS;
TEST_P(ContingencyMatrixTestS, Result) { RunTest(); }
INSTANTIATE_TEST_CASE_P(ContingencyMatrix, ContingencyMatrixTestS, ::testing::ValuesIn(inputs));
}  // namespace Metrics
}  // namespace MLCommon
