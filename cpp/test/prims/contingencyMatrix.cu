#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <algorithm>
#include <iostream>
#include <random>
#include "metrics/contingencyMatrix.h"
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

struct contingencyMatrixParam {
  int nElements;
  int minClass;
  int maxClass;
  bool calcCardinality;
  bool skipLabels;
  float tolerance;
};

template <typename T>
class ContingencyMatrixTestImpl
  : public ::testing::TestWithParam<contingencyMatrixParam> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<contingencyMatrixParam>::GetParam();

    int numElements = params.nElements;
    int lowerLabelRange = params.minClass;
    int upperLabelRange = params.maxClass;

    std::vector<int> y(numElements, 0);
    std::vector<int> y_hat(numElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_int_distribution<int> intGenerator(lowerLabelRange,
                                                    upperLabelRange);

    std::generate(y.begin(), y.end(), [&]() { return intGenerator(dre); });
    std::generate(y_hat.begin(), y_hat.end(),
                  [&]() { return intGenerator(dre); });

    if (params.skipLabels) {
      // remove two label value from input arrays
      int y1 = (upperLabelRange - lowerLabelRange) / 2;
      int y2 = y1 + (upperLabelRange - lowerLabelRange) / 4;

      // replacement values
      int y1_R = y1 + 1;
      int y2_R = y2 + 1;

      std::replace(y.begin(), y.end(), y1, y1_R);
      std::replace(y.begin(), y.end(), y2, y2_R);
      std::replace(y_hat.begin(), y_hat.end(), y1, y1_R);
      std::replace(y_hat.begin(), y_hat.end(), y2, y2_R);
    }

    numUniqueClasses = upperLabelRange - lowerLabelRange + 1;

    // generate golden output on CPU
    size_t sizeOfMat = numUniqueClasses * numUniqueClasses * sizeof(int);
    int *hGoldenOutput = (int *)malloc(sizeOfMat);
    memset(hGoldenOutput, 0, sizeOfMat);

    for (int i = 0; i < numElements; i++) {
      int row = y[i] - lowerLabelRange;
      int column = y_hat[i] - lowerLabelRange;

      hGoldenOutput[row * numUniqueClasses + column] += 1;
    }

    CUDA_CHECK(hipStreamCreate(&stream));
    MLCommon::allocate(dY, numElements);
    MLCommon::allocate(dYHat, numElements);
    MLCommon::allocate(dComputedOutput, numUniqueClasses * numUniqueClasses);
    MLCommon::allocate(dGoldenOutput, numUniqueClasses * numUniqueClasses);

    size_t workspaceSz = MLCommon::Metrics::getContingencyMatrixWorkspaceSize(
      numElements, dY, stream, lowerLabelRange, upperLabelRange);

    if (workspaceSz != 0) MLCommon::allocate(pWorkspace, workspaceSz);

    MLCommon::updateDevice(dYHat, &y_hat[0], numElements, stream);
    MLCommon::updateDevice(dY, &y[0], numElements, stream);
    MLCommon::updateDevice(dGoldenOutput, hGoldenOutput,
                           numUniqueClasses * numUniqueClasses, stream);

    if (params.calcCardinality) {
      T minLabel, maxLabel;
      MLCommon::Metrics::getInputClassCardinality(dY, numElements, stream,
                                                  minLabel, maxLabel);
      // allocate dComputedOutput using minLabel, maxLabel count - already done above
      MLCommon::Metrics::contingencyMatrix(
        dY, dYHat, numElements, dComputedOutput, stream, (void *)pWorkspace,
        workspaceSz, minLabel, maxLabel);
    } else
      MLCommon::Metrics::contingencyMatrix(
        dY, dYHat, numElements, dComputedOutput, stream, (void *)pWorkspace,
        workspaceSz, lowerLabelRange, upperLabelRange);
  }

  void TearDown() override {
    free(hGoldenOutput);
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(dY));
    CUDA_CHECK(hipFree(dYHat));
    CUDA_CHECK(hipFree(dComputedOutput));
    CUDA_CHECK(hipFree(dGoldenOutput));
    if (pWorkspace) CUDA_CHECK(hipFree(pWorkspace));
  }

  contingencyMatrixParam params;
  int numUniqueClasses = -1;
  T *dY = nullptr;
  T *dYHat = nullptr;
  int *dComputedOutput = nullptr;
  int *dGoldenOutput = nullptr;
  int *hGoldenOutput = nullptr;
  char *pWorkspace = nullptr;
  hipStream_t stream;
};

const std::vector<contingencyMatrixParam> inputs = {
  {10000, 1, 10, true, false, 0.000001},
  {100000, 1, 100, false, false, 0.000001},
  {1000000, 1, 1200, true, false, 0.000001},
  {1000000, 1, 10000, false, false, 0.000001},
  {100000, 1, 100, false, true, 0.000001}};

typedef ContingencyMatrixTestImpl<int> ContingencyMatrixTestImplS;
TEST_P(ContingencyMatrixTestImplS, Result) {
  ASSERT_TRUE(devArrMatch(dComputedOutput, dGoldenOutput,
                          numUniqueClasses * numUniqueClasses,
                          CompareApprox<float>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(ContingencyMatrix, ContingencyMatrixTestImplS,
                        ::testing::ValuesIn(inputs));
}  // namespace Metrics
}  // namespace MLCommon
