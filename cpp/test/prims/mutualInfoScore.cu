#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>
#include <algorithm>
#include <cuml/common/cuml_allocator.hpp>
#include <iostream>
#include <random>
#include "metrics/contingencyMatrix.h"
#include "metrics/mutualInfoScore.h"
#include "test_utils.h"

namespace MLCommon {
namespace Metrics {

//parameter structure definition
struct mutualInfoParam {
  int nElements;
  int lowerLabelRange;
  int upperLabelRange;
  bool sameArrays;
  double tolerance;
};

//test fixture class
template <typename T>
class mutualInfoTest : public ::testing::TestWithParam<mutualInfoParam> {
 protected:
  //the constructor
  void SetUp() override {
    //getting the parameters
    params = ::testing::TestWithParam<mutualInfoParam>::GetParam();

    nElements = params.nElements;
    lowerLabelRange = params.lowerLabelRange;
    upperLabelRange = params.upperLabelRange;

    //generating random value test input
    std::vector<int> arr1(nElements, 0);
    std::vector<int> arr2(nElements, 0);
    std::random_device rd;
    std::default_random_engine dre(rd());
    std::uniform_int_distribution<int> intGenerator(lowerLabelRange,
                                                    upperLabelRange);

    std::generate(arr1.begin(), arr1.end(),
                  [&]() { return intGenerator(dre); });
    if (params.sameArrays) {
      arr2 = arr1;
    } else {
      std::generate(arr2.begin(), arr2.end(),
                    [&]() { return intGenerator(dre); });
    }

    //generating the golden output
    //calculating the contingency matrix
    int numUniqueClasses = upperLabelRange - lowerLabelRange + 1;
    size_t sizeOfMat = numUniqueClasses * numUniqueClasses * sizeof(int);
    int *hGoldenOutput = (int *)malloc(sizeOfMat);
    memset(hGoldenOutput, 0, sizeOfMat);
    int i, j;
    for (i = 0; i < nElements; i++) {
      int row = arr1[i] - lowerLabelRange;
      int column = arr2[i] - lowerLabelRange;

      hGoldenOutput[row * numUniqueClasses + column] += 1;
    }

    int *a = (int *)malloc(numUniqueClasses * sizeof(int));
    int *b = (int *)malloc(numUniqueClasses * sizeof(int));
    memset(a, 0, numUniqueClasses * sizeof(int));
    memset(b, 0, numUniqueClasses * sizeof(int));

    //and also the reducing contingency matrix along row and column
    for (i = 0; i < numUniqueClasses; ++i) {
      for (j = 0; j < numUniqueClasses; ++j) {
        a[i] += hGoldenOutput[i * numUniqueClasses + j];
        b[i] += hGoldenOutput[j * numUniqueClasses + i];
      }
    }

    //calculating the truth mutual information
    for (int i = 0; i < numUniqueClasses; ++i) {
      for (int j = 0; j < numUniqueClasses; ++j) {
        if (a[i] * b[j] != 0 && hGoldenOutput[i * numUniqueClasses + j] != 0) {
          truthmutualInfo +=
            (double)(hGoldenOutput[i * numUniqueClasses + j]) *
            (log((double)(double(nElements) *
                          hGoldenOutput[i * numUniqueClasses + j])) -
             log((double)(a[i] * b[j])));
        }
      }
    }

    truthmutualInfo /= nElements;

    //allocating and initializing memory to the GPU
    CUDA_CHECK(hipStreamCreate(&stream));
    MLCommon::allocate(firstClusterArray, nElements, true);
    MLCommon::allocate(secondClusterArray, nElements, true);

    MLCommon::updateDevice(firstClusterArray, &arr1[0], (int)nElements, stream);
    MLCommon::updateDevice(secondClusterArray, &arr2[0], (int)nElements,
                           stream);
    std::shared_ptr<MLCommon::deviceAllocator> allocator(
      new defaultDeviceAllocator);

    //calling the mutualInfo CUDA implementation
    computedmutualInfo = MLCommon::Metrics::mutualInfoScore(
      firstClusterArray, secondClusterArray, nElements, lowerLabelRange,
      upperLabelRange, allocator, stream);
  }

  //the destructor
  void TearDown() override {
    CUDA_CHECK(hipFree(firstClusterArray));
    CUDA_CHECK(hipFree(secondClusterArray));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  //declaring the data values
  mutualInfoParam params;
  T lowerLabelRange, upperLabelRange;
  T *firstClusterArray = nullptr;
  T *secondClusterArray = nullptr;
  int nElements = 0;
  double truthmutualInfo = 0;
  double computedmutualInfo = 0;
  hipStream_t stream;
};

//setting test parameter values
const std::vector<mutualInfoParam> inputs = {
  {199, 1, 10, false, 0.000001},  {200, 15, 100, false, 0.000001},
  {100, 1, 20, false, 0.000001},  {10, 1, 10, false, 0.000001},
  {198, 1, 100, false, 0.000001}, {300, 3, 99, false, 0.000001},
  {199, 1, 10, true, 0.000001},   {200, 15, 100, true, 0.000001},
  {100, 1, 20, true, 0.000001},   {10, 1, 10, true, 0.000001},
  {198, 1, 100, true, 0.000001},  {300, 3, 99, true, 0.000001}};

//writing the test suite
typedef mutualInfoTest<int> mutualInfoTestClass;
TEST_P(mutualInfoTestClass, Result) {
  ASSERT_NEAR(computedmutualInfo, truthmutualInfo, params.tolerance);
}
INSTANTIATE_TEST_CASE_P(mutualInfo, mutualInfoTestClass,
                        ::testing::ValuesIn(inputs));

}  //end namespace Metrics
}  //end namespace MLCommon
