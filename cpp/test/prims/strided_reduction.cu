/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/strided_reduction.h"
#include "random/rng.h"
#include "reduce.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

template <typename T>
struct stridedReductionInputs {
  T tolerance;
  int rows, cols;
  unsigned long long int seed;
};

template <typename T>
void stridedReductionLaunch(T *dots, const T *data, int cols, int rows,
                            hipStream_t stream) {
  stridedReduction(dots, data, cols, rows, (T)0, stream, false,
                   [] __device__(T in, int i) { return in * in; });
}

template <typename T>
class stridedReductionTest
  : public ::testing::TestWithParam<stridedReductionInputs<T>> {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    params = ::testing::TestWithParam<stridedReductionInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;

    allocate(data, len);
    allocate(dots_exp, cols);  //expected dot products (from test)
    allocate(dots_act, cols);  //actual dot products (from prim)
    r.uniform(data, len, T(-1.0), T(1.0),
              stream);  //initialize matrix to random

    unaryAndGemv(dots_exp, data, cols, rows, stream);
    stridedReductionLaunch(dots_act, data, cols, rows, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(dots_exp));
    CUDA_CHECK(hipFree(dots_act));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  stridedReductionInputs<T> params;
  T *data, *dots_exp, *dots_act;
  hipStream_t stream;
};

const std::vector<stridedReductionInputs<float>> inputsf = {
  {0.00001f, 1024, 32, 1234ULL},
  {0.00001f, 1024, 64, 1234ULL},
  {0.00001f, 1024, 128, 1234ULL},
  {0.00001f, 1024, 256, 1234ULL}};

const std::vector<stridedReductionInputs<double>> inputsd = {
  {0.000000001, 1024, 32, 1234ULL},
  {0.000000001, 1024, 64, 1234ULL},
  {0.000000001, 1024, 128, 1234ULL},
  {0.000000001, 1024, 256, 1234ULL}};

typedef stridedReductionTest<float> stridedReductionTestF;
TEST_P(stridedReductionTestF, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<float>(params.tolerance)));
}

typedef stridedReductionTest<double> stridedReductionTestD;
TEST_P(stridedReductionTestD, Result) {
  ASSERT_TRUE(devArrMatch(dots_exp, dots_act, params.cols,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(stridedReductionTests, stridedReductionTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(stridedReductionTests, stridedReductionTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace LinAlg
}  // end namespace MLCommon
