#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "linalg/eltwise.h"
#include "random/rng.h"
#include "test_utils.h"

namespace MLCommon {
namespace LinAlg {

//// Testing unary ops

template <typename Type>
__global__ void naiveScaleKernel(Type *out, const Type *in, Type scalar,
                                 int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = scalar * in[idx];
  }
}

template <typename Type>
void naiveScale(Type *out, const Type *in, Type scalar, int len,
                hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naiveScaleKernel<Type><<<nblks, TPB, 0, stream>>>(out, in, scalar, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct ScalarMultiplyInputs {
  T tolerance;
  int len;
  T scalar;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os,
                           const ScalarMultiplyInputs<T> &dims) {
  return os;
}

template <typename T>
class ScalarMultiplyTest
  : public ::testing::TestWithParam<ScalarMultiplyInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<ScalarMultiplyInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    int len = params.len;
    T scalar = params.scalar;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in, len, T(-1.0), T(1.0), stream);
    naiveScale(out_ref, in, scalar, len, stream);
    scalarMultiply(out, in, scalar, len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

 protected:
  ScalarMultiplyInputs<T> params;
  T *in, *out_ref, *out;
};

const std::vector<ScalarMultiplyInputs<float>> inputsf1 = {
  {0.000001f, 1024 * 1024, 2.f, 1234ULL}};

const std::vector<ScalarMultiplyInputs<double>> inputsd1 = {
  {0.00000001, 1024 * 1024, 2.0, 1234ULL}};

typedef ScalarMultiplyTest<float> ScalarMultiplyTestF;
TEST_P(ScalarMultiplyTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}

typedef ScalarMultiplyTest<double> ScalarMultiplyTestD;
TEST_P(ScalarMultiplyTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(ScalarMultiplyTests, ScalarMultiplyTestF,
                        ::testing::ValuesIn(inputsf1));

INSTANTIATE_TEST_CASE_P(ScalarMultiplyTests, ScalarMultiplyTestD,
                        ::testing::ValuesIn(inputsd1));

//// Testing binary ops

template <typename Type>
__global__ void naiveAddKernel(Type *out, const Type *in1, const Type *in2,
                               int len) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    out[idx] = in1[idx] + in2[idx];
  }
}

template <typename Type>
void naiveAdd(Type *out, const Type *in1, const Type *in2, int len,
              hipStream_t stream) {
  static const int TPB = 64;
  int nblks = ceildiv(len, TPB);
  naiveAddKernel<Type><<<nblks, TPB, 0, stream>>>(out, in1, in2, len);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct EltwiseAddInputs {
  T tolerance;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os,
                           const EltwiseAddInputs<T> &dims) {
  return os;
}

template <typename T>
class EltwiseAddTest : public ::testing::TestWithParam<EltwiseAddInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<EltwiseAddInputs<T>>::GetParam();
    Random::Rng r(params.seed);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    int len = params.len;
    allocate(in1, len);
    allocate(in2, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in1, len, T(-1.0), T(1.0), stream);
    r.uniform(in2, len, T(-1.0), T(1.0), stream);
    naiveAdd(out_ref, in1, in2, len, stream);
    eltwiseAdd(out, in1, in2, len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

 protected:
  EltwiseAddInputs<T> params;
  T *in1, *in2, *out_ref, *out;
};

const std::vector<EltwiseAddInputs<float>> inputsf2 = {
  {0.000001f, 1024 * 1024, 1234ULL}};

const std::vector<EltwiseAddInputs<double>> inputsd2 = {
  {0.00000001, 1024 * 1024, 1234ULL}};

typedef EltwiseAddTest<float> EltwiseAddTestF;
TEST_P(EltwiseAddTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}

typedef EltwiseAddTest<double> EltwiseAddTestD;
TEST_P(EltwiseAddTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(EltwiseAddTests, EltwiseAddTestF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(EltwiseAddTests, EltwiseAddTestD,
                        ::testing::ValuesIn(inputsd2));

}  // end namespace LinAlg
}  // end namespace MLCommon
