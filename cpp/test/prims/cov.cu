/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include "random/rng.h"
#include "stats/cov.h"
#include "stats/mean.h"
#include "test_utils.h"

namespace MLCommon {
namespace Stats {

template <typename T>
struct CovInputs {
  T tolerance, mean, var;
  int rows, cols;
  bool sample, rowMajor, stable;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const CovInputs<T> &dims) {
  return os;
}

template <typename T>
class CovTest : public ::testing::TestWithParam<CovInputs<T>> {
 protected:
  void SetUp() override {
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));
    params = ::testing::TestWithParam<CovInputs<T>>::GetParam();
    params.tolerance *= 2;
    Random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;
    T var = params.var;
    allocate(data, len);
    allocate(mean_act, cols);
    allocate(cov_act, cols * cols);
    r.normal(data, len, params.mean, var, stream);
    mean(mean_act, data, cols, rows, params.sample, params.rowMajor, stream);
    cov(cov_act, data, mean_act, cols, rows, params.sample, params.rowMajor,
        params.stable, handle, stream);

    T data_h[6] = {1.0, 2.0, 5.0, 4.0, 2.0, 1.0};
    T cov_cm_ref_h[4] = {4.3333, -2.8333, -2.8333, 2.333};

    allocate(data_cm, 6);
    allocate(cov_cm, 4);
    allocate(cov_cm_ref, 4);
    allocate(mean_cm, 2);

    updateDevice(data_cm, data_h, 6, stream);
    updateDevice(cov_cm_ref, cov_cm_ref_h, 4, stream);

    mean(mean_cm, data_cm, 2, 3, true, false, stream);
    cov(cov_cm, data_cm, mean_cm, 2, 3, true, false, true, handle, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(mean_act));
    CUDA_CHECK(hipFree(cov_act));
    CUDA_CHECK(hipFree(data_cm));
    CUDA_CHECK(hipFree(cov_cm));
    CUDA_CHECK(hipFree(cov_cm_ref));
    CUDA_CHECK(hipFree(mean_cm));
    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  CovInputs<T> params;
  T *data, *mean_act, *cov_act;
  hipblasHandle_t handle;
  hipStream_t stream;

  T *data_cm, *cov_cm, *cov_cm_ref, *mean_cm;
};

///@todo: add stable=false after it has been implemented
const std::vector<CovInputs<float>> inputsf = {
  {0.03f, 1.f, 2.f, 32 * 1024, 32, true, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 64, true, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 128, true, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 256, true, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 32, false, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 64, false, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 128, false, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 256, false, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 32, true, true, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 64, true, true, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 128, true, true, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 256, true, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 32, false, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 64, false, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 128, false, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 256, false, true, true, 1234ULL}};

const std::vector<CovInputs<double>> inputsd = {
  {0.03, 1.0, 2.0, 32 * 1024, 32, true, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 64, true, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 128, true, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 256, true, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 32, false, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 64, false, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 128, false, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 256, false, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 32, true, true, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 64, true, true, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 128, true, true, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 256, true, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 32, false, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 64, false, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 128, false, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 256, false, true, true, 1234ULL}};

typedef CovTest<float> CovTestF;
TEST_P(CovTestF, Result) {
  ASSERT_TRUE(diagonalMatch(params.var * params.var, cov_act, params.cols,
                            params.cols,
                            CompareApprox<float>(params.tolerance)));
}

typedef CovTest<double> CovTestD;
TEST_P(CovTestD, Result) {
  ASSERT_TRUE(diagonalMatch(params.var * params.var, cov_act, params.cols,
                            params.cols,
                            CompareApprox<double>(params.tolerance)));
}

typedef CovTest<float> CovTestSmallF;
TEST_P(CovTestSmallF, Result) {
  ASSERT_TRUE(devArrMatch(cov_cm_ref, cov_cm, 2, 2,
                          CompareApprox<float>(params.tolerance)));
}

typedef CovTest<double> CovTestSmallD;
TEST_P(CovTestSmallD, Result) {
  ASSERT_TRUE(devArrMatch(cov_cm_ref, cov_cm, 2, 2,
                          CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(CovTests, CovTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(CovTests, CovTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_CASE_P(CovTests, CovTestSmallF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(CovTests, CovTestSmallD, ::testing::ValuesIn(inputsd));

}  // end namespace Stats
}  // end namespace MLCommon
