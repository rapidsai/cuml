/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/random/rng.cuh>
#include <raft/stats/mean.cuh>
#include <stats/cov.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Stats {

template <typename T>
struct CovInputs {
  T tolerance, mean, var;
  int rows, cols;
  bool sample, rowMajor, stable;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const CovInputs<T> &dims) {
  return os;
}

template <typename T>
class CovTest : public ::testing::TestWithParam<CovInputs<T>> {
 protected:
  void SetUp() override {
    raft::handle_t handle;
    hipStream_t stream = handle.get_stream();

    params = ::testing::TestWithParam<CovInputs<T>>::GetParam();
    params.tolerance *= 2;
    raft::random::Rng r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;
    T var = params.var;
    raft::allocate(data, len);
    raft::allocate(mean_act, cols);
    raft::allocate(cov_act, cols * cols);
    r.normal(data, len, params.mean, var, stream);
    raft::stats::mean(mean_act, data, cols, rows, params.sample,
                      params.rowMajor, stream);
    cov(handle, cov_act, data, mean_act, cols, rows, params.sample,
        params.rowMajor, params.stable, stream);

    T data_h[6] = {1.0, 2.0, 5.0, 4.0, 2.0, 1.0};
    T cov_cm_ref_h[4] = {4.3333, -2.8333, -2.8333, 2.333};

    raft::allocate(data_cm, 6);
    raft::allocate(cov_cm, 4);
    raft::allocate(cov_cm_ref, 4);
    raft::allocate(mean_cm, 2);

    raft::update_device(data_cm, data_h, 6, stream);
    raft::update_device(cov_cm_ref, cov_cm_ref_h, 4, stream);

    raft::stats::mean(mean_cm, data_cm, 2, 3, true, false, stream);
    cov(handle, cov_cm, data_cm, mean_cm, 2, 3, true, false, true, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(mean_act));
    CUDA_CHECK(hipFree(cov_act));
    CUDA_CHECK(hipFree(data_cm));
    CUDA_CHECK(hipFree(cov_cm));
    CUDA_CHECK(hipFree(cov_cm_ref));
    CUDA_CHECK(hipFree(mean_cm));
  }

 protected:
  CovInputs<T> params;
  T *data, *mean_act, *cov_act;
  hipblasHandle_t handle;
  hipStream_t stream;

  T *data_cm, *cov_cm, *cov_cm_ref, *mean_cm;
};

///@todo: add stable=false after it has been implemented
const std::vector<CovInputs<float>> inputsf = {
  {0.03f, 1.f, 2.f, 32 * 1024, 32, true, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 64, true, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 128, true, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 256, true, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 32, false, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 64, false, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 128, false, false, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 256, false, false, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 32, true, true, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 64, true, true, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 128, true, true, true, 1234ULL},
  {0.03f, 1.f, 2.f, 32 * 1024, 256, true, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 32, false, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 64, false, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 128, false, true, true, 1234ULL},
  {0.03f, -1.f, 2.f, 32 * 1024, 256, false, true, true, 1234ULL}};

const std::vector<CovInputs<double>> inputsd = {
  {0.03, 1.0, 2.0, 32 * 1024, 32, true, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 64, true, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 128, true, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 256, true, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 32, false, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 64, false, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 128, false, false, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 256, false, false, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 32, true, true, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 64, true, true, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 128, true, true, true, 1234ULL},
  {0.03, 1.0, 2.0, 32 * 1024, 256, true, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 32, false, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 64, false, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 128, false, true, true, 1234ULL},
  {0.03, -1.0, 2.0, 32 * 1024, 256, false, true, true, 1234ULL}};

typedef CovTest<float> CovTestF;
TEST_P(CovTestF, Result) {
  ASSERT_TRUE(raft::diagonalMatch(
    params.var * params.var, cov_act, params.cols, params.cols,
    raft::CompareApprox<float>(params.tolerance)));
}

typedef CovTest<double> CovTestD;
TEST_P(CovTestD, Result) {
  ASSERT_TRUE(raft::diagonalMatch(
    params.var * params.var, cov_act, params.cols, params.cols,
    raft::CompareApprox<double>(params.tolerance)));
}

typedef CovTest<float> CovTestSmallF;
TEST_P(CovTestSmallF, Result) {
  ASSERT_TRUE(raft::devArrMatch(cov_cm_ref, cov_cm, 2, 2,
                                raft::CompareApprox<float>(params.tolerance)));
}

typedef CovTest<double> CovTestSmallD;
TEST_P(CovTestSmallD, Result) {
  ASSERT_TRUE(raft::devArrMatch(cov_cm_ref, cov_cm, 2, 2,
                                raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(CovTests, CovTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(CovTests, CovTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_CASE_P(CovTests, CovTestSmallF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(CovTests, CovTestSmallD, ::testing::ValuesIn(inputsd));

}  // end namespace Stats
}  // end namespace MLCommon
