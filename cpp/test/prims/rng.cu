#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <hipcub/hipcub.hpp>
#include "cuda_utils.h"
#include "random/rng.h"
#include "stats/mean.h"
#include "stats/stddev.h"
#include "test_utils.h"

namespace MLCommon {
namespace Random {

enum RandomType {
  RNG_Normal,
  RNG_LogNormal,
  RNG_Uniform,
  RNG_Gumbel,
  RNG_Logistic,
  RNG_Exp,
  RNG_Rayleigh,
  RNG_Laplace
};

template <typename T, int TPB>
__global__ void meanKernel(T* out, const T* data, int len) {
  typedef hipcub::BlockReduce<T, TPB> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  T val = tid < len ? data[tid] : T(0);
  T x = BlockReduce(temp_storage).Sum(val);
  __syncthreads();
  T xx = BlockReduce(temp_storage).Sum(val * val);
  __syncthreads();
  if (threadIdx.x == 0) {
    myAtomicAdd(out, x);
    myAtomicAdd(out + 1, xx);
  }
}

template <typename T>
struct RngInputs {
  T tolerance;
  int len;
  // start, end: for uniform
  // mean, sigma: for normal/lognormal
  // mean, beta: for gumbel
  // mean, scale: for logistic and laplace
  // lambda: for exponential
  // sigma: for rayleigh
  T start, end;
  RandomType type;
  GeneratorType gtype;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const RngInputs<T>& dims) {
  return os;
}

#include <sys/timeb.h>
#include <time.h>

template <typename T>
class RngTest : public ::testing::TestWithParam<RngInputs<T>> {
 protected:
  void SetUp() override {
    // Tests are configured with their expected test-values sigma. For example,
    // 4 x sigma indicates the test shouldn't fail 99.9% of the time.
    num_sigma = 10;
    params = ::testing::TestWithParam<RngInputs<T>>::GetParam();
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    Rng r(params.seed, params.gtype);
    allocate(data, params.len);
    allocate(stats, 2, true);
    switch (params.type) {
      case RNG_Normal:
        r.normal(data, params.len, params.start, params.end, stream);
        break;
      case RNG_LogNormal:
        r.lognormal(data, params.len, params.start, params.end, stream);
        break;
      case RNG_Uniform:
        r.uniform(data, params.len, params.start, params.end, stream);
        break;
      case RNG_Gumbel:
        r.gumbel(data, params.len, params.start, params.end, stream);
        break;
      case RNG_Logistic:
        r.logistic(data, params.len, params.start, params.end, stream);
        break;
      case RNG_Exp:
        r.exponential(data, params.len, params.start, stream);
        break;
      case RNG_Rayleigh:
        r.rayleigh(data, params.len, params.start, stream);
        break;
      case RNG_Laplace:
        r.laplace(data, params.len, params.start, params.end, stream);
        break;
    };
    static const int threads = 128;
    meanKernel<T, threads>
      <<<ceildiv(params.len, threads), threads, 0, stream>>>(stats, data,
                                                             params.len);
    updateHost<T>(h_stats, stats, 2, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    h_stats[0] /= params.len;
    h_stats[1] = (h_stats[1] / params.len) - (h_stats[0] * h_stats[0]);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(stats));
  }

  void getExpectedMeanVar(T meanvar[2]) {
    switch (params.type) {
      case RNG_Normal:
        meanvar[0] = params.start;
        meanvar[1] = params.end * params.end;
        break;
      case RNG_LogNormal: {
        auto var = params.end * params.end;
        auto mu = params.start;
        meanvar[0] = myExp(mu + var * T(0.5));
        meanvar[1] = (myExp(var) - T(1.0)) * myExp(T(2.0) * mu + var);
        break;
      }
      case RNG_Uniform:
        meanvar[0] = (params.start + params.end) * T(0.5);
        meanvar[1] = params.end - params.start;
        meanvar[1] = meanvar[1] * meanvar[1] / T(12.0);
        break;
      case RNG_Gumbel: {
        auto gamma = T(0.577215664901532);
        meanvar[0] = params.start + params.end * gamma;
        meanvar[1] = T(3.1415) * T(3.1415) * params.end * params.end / T(6.0);
        break;
      }
      case RNG_Logistic:
        meanvar[0] = params.start;
        meanvar[1] = T(3.1415) * T(3.1415) * params.end * params.end / T(3.0);
        break;
      case RNG_Exp:
        meanvar[0] = T(1.0) / params.start;
        meanvar[1] = meanvar[0] * meanvar[0];
        break;
      case RNG_Rayleigh:
        meanvar[0] = params.start * mySqrt(T(3.1415 / 2.0));
        meanvar[1] =
          ((T(4.0) - T(3.1415)) / T(2.0)) * params.start * params.start;
        break;
      case RNG_Laplace:
        meanvar[0] = params.start;
        meanvar[1] = T(2.0) * params.end * params.end;
        break;
    };
  }

 protected:
  RngInputs<T> params;
  T *data, *stats;
  T h_stats[2];  // mean, var
  int num_sigma;
};

// The measured mean and standard deviation for each tested distribution are,
// of course, statistical variables. Thus setting an appropriate testing
// tolerance essentially requires one to set a probability of test failure. We
// choose to set this at 3-4 x sigma, i.e., a 99.7-99.9% confidence interval so that
// the test will indeed pass. In quick experiments (using the identical
// distributions given by NumPy/SciPy), the measured standard deviation is the
// variable with the greatest variance and so we determined the variance for
// each distribution and number of samples (32*1024 or 8*1024). Below
// are listed the standard deviation for these tests.

// Distribution: StdDev 32*1024, StdDev 8*1024
// Normal: 0.0055, 0.011
// LogNormal: 0.05, 0.1
// Uniform: 0.003, 0.005
// Gumbel: 0.005, 0.01
// Logistic: 0.005, 0.01
// Exp: 0.008, 0.015
// Rayleigh: 0.0125, 0.025
// Laplace: 0.02, 0.04

// We generally want 4 x sigma >= 99.9% chance of success

typedef RngTest<float> RngTestF;
const std::vector<RngInputs<float>> inputsf = {
  {0.0055, 32 * 1024, 1.f, 1.f, RNG_Normal, GenPhilox, 1234ULL},
  {0.011, 8 * 1024, 1.f, 1.f, RNG_Normal, GenPhilox, 1234ULL},
  {0.05, 32 * 1024, 1.f, 1.f, RNG_LogNormal, GenPhilox, 1234ULL},
  {0.1, 8 * 1024, 1.f, 1.f, RNG_LogNormal, GenPhilox, 1234ULL},
  {0.003, 32 * 1024, -1.f, 1.f, RNG_Uniform, GenPhilox, 1234ULL},
  {0.005, 8 * 1024, -1.f, 1.f, RNG_Uniform, GenPhilox, 1234ULL},
  {0.005, 32 * 1024, 1.f, 1.f, RNG_Gumbel, GenPhilox, 1234ULL},
  {0.01, 8 * 1024, 1.f, 1.f, RNG_Gumbel, GenPhilox, 1234ULL},
  {0.005, 32 * 1024, 1.f, 1.f, RNG_Logistic, GenPhilox, 1234ULL},
  {0.01, 8 * 1024, 1.f, 1.f, RNG_Logistic, GenPhilox, 1234ULL},
  {0.008, 32 * 1024, 1.f, 1.f, RNG_Exp, GenPhilox, 1234ULL},
  {0.015, 8 * 1024, 1.f, 1.f, RNG_Exp, GenPhilox, 1234ULL},
  {0.0125, 32 * 1024, 1.f, 1.f, RNG_Rayleigh, GenPhilox, 1234ULL},
  {0.025, 8 * 1024, 1.f, 1.f, RNG_Rayleigh, GenPhilox, 1234ULL},
  {0.02, 32 * 1024, 1.f, 1.f, RNG_Laplace, GenPhilox, 1234ULL},
  {0.04, 8 * 1024, 1.f, 1.f, RNG_Laplace, GenPhilox, 1234ULL},

  {0.0055, 32 * 1024, 1.f, 1.f, RNG_Normal, GenTaps, 1234ULL},
  {0.011, 8 * 1024, 1.f, 1.f, RNG_Normal, GenTaps, 1234ULL},
  {0.05, 32 * 1024, 1.f, 1.f, RNG_LogNormal, GenTaps, 1234ULL},
  {0.1, 8 * 1024, 1.f, 1.f, RNG_LogNormal, GenTaps, 1234ULL},
  {0.003, 32 * 1024, -1.f, 1.f, RNG_Uniform, GenTaps, 1234ULL},
  {0.005, 8 * 1024, -1.f, 1.f, RNG_Uniform, GenTaps, 1234ULL},
  {0.005, 32 * 1024, 1.f, 1.f, RNG_Gumbel, GenTaps, 1234ULL},
  {0.01, 8 * 1024, 1.f, 1.f, RNG_Gumbel, GenTaps, 1234ULL},
  {0.005, 32 * 1024, 1.f, 1.f, RNG_Logistic, GenTaps, 1234ULL},
  {0.01, 8 * 1024, 1.f, 1.f, RNG_Logistic, GenTaps, 1234ULL},
  {0.008, 32 * 1024, 1.f, 1.f, RNG_Exp, GenTaps, 1234ULL},
  {0.015, 8 * 1024, 1.f, 1.f, RNG_Exp, GenTaps, 1234ULL},
  {0.0125, 32 * 1024, 1.f, 1.f, RNG_Rayleigh, GenTaps, 1234ULL},
  {0.025, 8 * 1024, 1.f, 1.f, RNG_Rayleigh, GenTaps, 1234ULL},
  {0.02, 32 * 1024, 1.f, 1.f, RNG_Laplace, GenTaps, 1234ULL},
  {0.04, 8 * 1024, 1.f, 1.f, RNG_Laplace, GenTaps, 1234ULL},

  {0.0055, 32 * 1024, 1.f, 1.f, RNG_Normal, GenKiss99, 1234ULL},
  {0.011, 8 * 1024, 1.f, 1.f, RNG_Normal, GenKiss99, 1234ULL},
  {0.05, 32 * 1024, 1.f, 1.f, RNG_LogNormal, GenKiss99, 1234ULL},
  {0.1, 8 * 1024, 1.f, 1.f, RNG_LogNormal, GenKiss99, 1234ULL},
  {0.003, 32 * 1024, -1.f, 1.f, RNG_Uniform, GenKiss99, 1234ULL},
  {0.005, 8 * 1024, -1.f, 1.f, RNG_Uniform, GenKiss99, 1234ULL},
  {0.005, 32 * 1024, 1.f, 1.f, RNG_Gumbel, GenKiss99, 1234ULL},
  {0.01, 8 * 1024, 1.f, 1.f, RNG_Gumbel, GenKiss99, 1234ULL},
  {0.005, 32 * 1024, 1.f, 1.f, RNG_Logistic, GenKiss99, 1234ULL},
  {0.01, 8 * 1024, 1.f, 1.f, RNG_Logistic, GenKiss99, 1234ULL},
  {0.008, 32 * 1024, 1.f, 1.f, RNG_Exp, GenKiss99, 1234ULL},
  {0.015, 8 * 1024, 1.f, 1.f, RNG_Exp, GenKiss99, 1234ULL},
  {0.0125, 32 * 1024, 1.f, 1.f, RNG_Rayleigh, GenKiss99, 1234ULL},
  {0.025, 8 * 1024, 1.f, 1.f, RNG_Rayleigh, GenKiss99, 1234ULL},
  {0.02, 32 * 1024, 1.f, 1.f, RNG_Laplace, GenKiss99, 1234ULL},
  {0.04, 8 * 1024, 1.f, 1.f, RNG_Laplace, GenKiss99, 1234ULL}};

TEST_P(RngTestF, Result) {
  float meanvar[2];
  getExpectedMeanVar(meanvar);
  ASSERT_TRUE(match(meanvar[0], h_stats[0],
                    CompareApprox<float>(num_sigma * params.tolerance)));
  ASSERT_TRUE(match(meanvar[1], h_stats[1],
                    CompareApprox<float>(num_sigma * params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(RngTests, RngTestF, ::testing::ValuesIn(inputsf));

typedef RngTest<double> RngTestD;
const std::vector<RngInputs<double>> inputsd = {
  {0.0055, 32 * 1024, 1.0, 1.0, RNG_Normal, GenPhilox, 1234ULL},
  {0.011, 8 * 1024, 1.0, 1.0, RNG_Normal, GenPhilox, 1234ULL},
  {0.05, 32 * 1024, 1.0, 1.0, RNG_LogNormal, GenPhilox, 1234ULL},
  {0.1, 8 * 1024, 1.0, 1.0, RNG_LogNormal, GenPhilox, 1234ULL},
  {0.003, 32 * 1024, -1.0, 1.0, RNG_Uniform, GenPhilox, 1234ULL},
  {0.005, 8 * 1024, -1.0, 1.0, RNG_Uniform, GenPhilox, 1234ULL},
  {0.005, 32 * 1024, 1.0, 1.0, RNG_Gumbel, GenPhilox, 1234ULL},
  {0.01, 8 * 1024, 1.0, 1.0, RNG_Gumbel, GenPhilox, 1234ULL},
  {0.005, 32 * 1024, 1.0, 1.0, RNG_Logistic, GenPhilox, 1234ULL},
  {0.01, 8 * 1024, 1.0, 1.0, RNG_Logistic, GenPhilox, 1234ULL},
  {0.008, 32 * 1024, 1.0, 1.0, RNG_Exp, GenPhilox, 1234ULL},
  {0.015, 8 * 1024, 1.0, 1.0, RNG_Exp, GenPhilox, 1234ULL},
  {0.0125, 32 * 1024, 1.0, 1.0, RNG_Rayleigh, GenPhilox, 1234ULL},
  {0.025, 8 * 1024, 1.0, 1.0, RNG_Rayleigh, GenPhilox, 1234ULL},
  {0.02, 32 * 1024, 1.0, 1.0, RNG_Laplace, GenPhilox, 1234ULL},
  {0.04, 8 * 1024, 1.0, 1.0, RNG_Laplace, GenPhilox, 1234ULL},

  {0.0055, 32 * 1024, 1.0, 1.0, RNG_Normal, GenTaps, 1234ULL},
  {0.011, 8 * 1024, 1.0, 1.0, RNG_Normal, GenTaps, 1234ULL},
  {0.05, 32 * 1024, 1.0, 1.0, RNG_LogNormal, GenTaps, 1234ULL},
  {0.1, 8 * 1024, 1.0, 1.0, RNG_LogNormal, GenTaps, 1234ULL},
  {0.003, 32 * 1024, -1.0, 1.0, RNG_Uniform, GenTaps, 1234ULL},
  {0.005, 8 * 1024, -1.0, 1.0, RNG_Uniform, GenTaps, 1234ULL},
  {0.005, 32 * 1024, 1.0, 1.0, RNG_Gumbel, GenTaps, 1234ULL},
  {0.01, 8 * 1024, 1.0, 1.0, RNG_Gumbel, GenTaps, 1234ULL},
  {0.005, 32 * 1024, 1.0, 1.0, RNG_Logistic, GenTaps, 1234ULL},
  {0.01, 8 * 1024, 1.0, 1.0, RNG_Logistic, GenTaps, 1234ULL},
  {0.008, 32 * 1024, 1.0, 1.0, RNG_Exp, GenTaps, 1234ULL},
  {0.015, 8 * 1024, 1.0, 1.0, RNG_Exp, GenTaps, 1234ULL},
  {0.0125, 32 * 1024, 1.0, 1.0, RNG_Rayleigh, GenTaps, 1234ULL},
  {0.025, 8 * 1024, 1.0, 1.0, RNG_Rayleigh, GenTaps, 1234ULL},
  {0.02, 32 * 1024, 1.0, 1.0, RNG_Laplace, GenTaps, 1234ULL},
  {0.04, 8 * 1024, 1.0, 1.0, RNG_Laplace, GenTaps, 1234ULL},

  {0.0055, 32 * 1024, 1.0, 1.0, RNG_Normal, GenKiss99, 1234ULL},
  {0.011, 8 * 1024, 1.0, 1.0, RNG_Normal, GenKiss99, 1234ULL},
  {0.05, 32 * 1024, 1.0, 1.0, RNG_LogNormal, GenKiss99, 1234ULL},
  {0.1, 8 * 1024, 1.0, 1.0, RNG_LogNormal, GenKiss99, 1234ULL},
  {0.003, 32 * 1024, -1.0, 1.0, RNG_Uniform, GenKiss99, 1234ULL},
  {0.005, 8 * 1024, -1.0, 1.0, RNG_Uniform, GenKiss99, 1234ULL},
  {0.005, 32 * 1024, 1.0, 1.0, RNG_Gumbel, GenKiss99, 1234ULL},
  {0.01, 8 * 1024, 1.0, 1.0, RNG_Gumbel, GenKiss99, 1234ULL},
  {0.005, 32 * 1024, 1.0, 1.0, RNG_Logistic, GenKiss99, 1234ULL},
  {0.01, 8 * 1024, 1.0, 1.0, RNG_Logistic, GenKiss99, 1234ULL},
  {0.008, 32 * 1024, 1.0, 1.0, RNG_Exp, GenKiss99, 1234ULL},
  {0.015, 8 * 1024, 1.0, 1.0, RNG_Exp, GenKiss99, 1234ULL},
  {0.0125, 32 * 1024, 1.0, 1.0, RNG_Rayleigh, GenKiss99, 1234ULL},
  {0.025, 8 * 1024, 1.0, 1.0, RNG_Rayleigh, GenKiss99, 1234ULL},
  {0.02, 32 * 1024, 1.0, 1.0, RNG_Laplace, GenKiss99, 1234ULL},
  {0.04, 8 * 1024, 1.0, 1.0, RNG_Laplace, GenKiss99, 1234ULL}};
TEST_P(RngTestD, Result) {
  double meanvar[2];
  getExpectedMeanVar(meanvar);
  ASSERT_TRUE(match(meanvar[0], h_stats[0],
                    CompareApprox<double>(num_sigma * params.tolerance)));
  ASSERT_TRUE(match(meanvar[1], h_stats[1],
                    CompareApprox<double>(num_sigma * params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(RngTests, RngTestD, ::testing::ValuesIn(inputsd));

// ---------------------------------------------------------------------- //
// Test for expected variance in mean calculations

template <typename T>
T quick_mean(const std::vector<T>& d) {
  T acc = T(0);
  for (const auto& di : d) {
    acc += di;
  }
  return acc / d.size();
}

template <typename T>
T quick_std(const std::vector<T>& d) {
  T acc = T(0);
  T d_mean = quick_mean(d);
  for (const auto& di : d) {
    acc += ((di - d_mean) * (di - d_mean));
  }
  return std::sqrt(acc / (d.size() - 1));
}

template <typename T>
std::ostream& operator<<(std::ostream& out, const std::vector<T>& v) {
  if (!v.empty()) {
    out << '[';
    std::copy(v.begin(), v.end(), std::ostream_iterator<T>(out, ", "));
    out << "\b\b]";
  }
  return out;
}

// The following tests the 3 random number generators by checking that the
// measured mean error is close to the well-known analytical result
// (sigma/sqrt(n_samples)). To compute the mean error, we a number of
// experiments computing the mean, giving us a distribution of the mean
// itself. The mean error is simply the standard deviation of this
// distribution (the standard deviation of the mean).
TEST(Rng, MeanError) {
  timeb time_struct;
  ftime(&time_struct);
  int seed = time_struct.millitm;
  int num_samples = 1024;
  int num_experiments = 1024;
  float* data;
  float* mean_result;
  float* std_result;
  int len = num_samples * num_experiments;

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  allocate(data, len);
  allocate(mean_result, num_experiments);
  allocate(std_result, num_experiments);

  for (auto rtype :
       {Random::GenPhilox, Random::GenKiss99 /*, Random::GenTaps */}) {
    Random::Rng r(seed, rtype);
    r.normal(data, len, 3.3f, 0.23f, stream);
    // r.uniform(data, len, -1.0, 2.0);
    Stats::mean(mean_result, data, num_samples, num_experiments, false, false,
                stream);
    Stats::stddev(std_result, data, mean_result, num_samples, num_experiments,
                  false, false, stream);
    std::vector<float> h_mean_result(num_experiments);
    std::vector<float> h_std_result(num_experiments);
    updateHost(h_mean_result.data(), mean_result, num_experiments, stream);
    updateHost(h_std_result.data(), std_result, num_experiments, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto d_mean = quick_mean(h_mean_result);

    // std-dev of mean; also known as mean error
    auto d_std_of_mean = quick_std(h_mean_result);
    auto d_std = quick_mean(h_std_result);
    auto d_std_of_mean_analytical = d_std / std::sqrt(num_samples);

    // std::cout << "measured mean error: " << d_std_of_mean << "\n";
    // std::cout << "expected mean error: " << d_std/std::sqrt(num_samples) << "\n";

    auto diff_expected_vs_measured_mean_error =
      std::abs(d_std_of_mean - d_std / std::sqrt(num_samples));

    ASSERT_TRUE(
      (diff_expected_vs_measured_mean_error / d_std_of_mean_analytical < 0.5));
  }
  CUDA_CHECK(hipStreamDestroy(stream));
  CUDA_CHECK(hipFree(data));
  CUDA_CHECK(hipFree(mean_result));
  CUDA_CHECK(hipFree(std_result));

  // std::cout << "mean_res:" << h_mean_result << "\n";
}

template <typename T, int len, int scale>
class ScaledBernoulliTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));

    Rng r(42);

    allocate(data, len * sizeof(T), stream);
    r.scaled_bernoulli(data, len, T(0.5), T(scale), stream);
  }

  void TearDown() override { CUDA_CHECK(hipFree(data)); }

  void rangeCheck() {
    T* h_data = new T[len];
    updateHost(h_data, data, len, stream);
    ASSERT_TRUE(std::none_of(h_data, h_data + len, [](const T& a) {
      return a < -scale || a > scale;
    }));
    delete[] h_data;
  }

  T* data;
  hipStream_t stream;
};

typedef ScaledBernoulliTest<float, 500, 35> ScaledBernoulliTest1;
TEST_F(ScaledBernoulliTest1, RangeCheck) { rangeCheck(); }

typedef ScaledBernoulliTest<double, 100, 220> ScaledBernoulliTest2;
TEST_F(ScaledBernoulliTest2, RangeCheck) { rangeCheck(); }

template <typename T, int len>
class BernoulliTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    Rng r(42);
    allocate(data, len * sizeof(bool), stream);
    r.bernoulli(data, len, T(0.5), stream);
  }

  void TearDown() override { CUDA_CHECK(hipFree(data)); }

  void trueFalseCheck() {
    // both true and false values must be present
    bool* h_data = new bool[len];
    updateHost(h_data, data, len, stream);
    ASSERT_TRUE(std::any_of(h_data, h_data + len, [](bool a) { return a; }));
    ASSERT_TRUE(std::any_of(h_data, h_data + len, [](bool a) { return !a; }));
    delete[] h_data;
  }

  bool* data;
  hipStream_t stream;
};

typedef BernoulliTest<float, 1000> BernoulliTest1;
TEST_F(BernoulliTest1, TrueFalseCheck) { trueFalseCheck(); }

typedef BernoulliTest<double, 1000> BernoulliTest2;
TEST_F(BernoulliTest2, TrueFalseCheck) { trueFalseCheck(); }

/** Rng::normalTable tests */
template <typename T>
struct RngNormalTableInputs {
  T tolerance;
  int rows, cols;
  T mu, sigma;
  GeneratorType gtype;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os,
                           const RngNormalTableInputs<T>& dims) {
  return os;
}

template <typename T>
class RngNormalTableTest
  : public ::testing::TestWithParam<RngNormalTableInputs<T>> {
 protected:
  void SetUp() override {
    // Tests are configured with their expected test-values sigma. For example,
    // 4 x sigma indicates the test shouldn't fail 99.9% of the time.
    num_sigma = 10;
    params = ::testing::TestWithParam<RngNormalTableInputs<T>>::GetParam();
    int len = params.rows * params.cols;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    Rng r(params.seed, params.gtype);
    allocate(data, len);
    allocate(stats, 2, true);
    allocate(mu_vec, params.cols);
    r.fill(mu_vec, params.cols, params.mu, stream);
    T* sigma_vec = nullptr;
    r.normalTable(data, params.rows, params.cols, mu_vec, sigma_vec,
                  params.sigma, stream);
    static const int threads = 128;
    meanKernel<T, threads>
      <<<ceildiv(len, threads), threads, 0, stream>>>(stats, data, len);
    updateHost<T>(h_stats, stats, 2, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    h_stats[0] /= len;
    h_stats[1] = (h_stats[1] / len) - (h_stats[0] * h_stats[0]);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(stats));
    CUDA_CHECK(hipFree(mu_vec));
  }

  void getExpectedMeanVar(T meanvar[2]) {
    meanvar[0] = params.mu;
    meanvar[1] = params.sigma * params.sigma;
  }

 protected:
  RngNormalTableInputs<T> params;
  T *data, *stats, *mu_vec;
  T h_stats[2];  // mean, var
  int num_sigma;
};

typedef RngNormalTableTest<float> RngNormalTableTestF;
const std::vector<RngNormalTableInputs<float>> inputsf_t = {
  {0.0055, 32, 1024, 1.f, 1.f, GenPhilox, 1234ULL},
  {0.011, 8, 1024, 1.f, 1.f, GenPhilox, 1234ULL},
  {0.0055, 32, 1024, 1.f, 1.f, GenTaps, 1234ULL},
  {0.011, 8, 1024, 1.f, 1.f, GenTaps, 1234ULL},
  {0.0055, 32, 1024, 1.f, 1.f, GenKiss99, 1234ULL},
  {0.011, 8, 1024, 1.f, 1.f, GenKiss99, 1234ULL}};

TEST_P(RngNormalTableTestF, Result) {
  float meanvar[2];
  getExpectedMeanVar(meanvar);
  ASSERT_TRUE(match(meanvar[0], h_stats[0],
                    CompareApprox<float>(num_sigma * params.tolerance)));
  ASSERT_TRUE(match(meanvar[1], h_stats[1],
                    CompareApprox<float>(num_sigma * params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(RngNormalTableTests, RngNormalTableTestF,
                        ::testing::ValuesIn(inputsf_t));

typedef RngNormalTableTest<double> RngNormalTableTestD;
const std::vector<RngNormalTableInputs<double>> inputsd_t = {
  {0.0055, 32, 1024, 1.0, 1.0, GenPhilox, 1234ULL},
  {0.011, 8, 1024, 1.0, 1.0, GenPhilox, 1234ULL},
  {0.0055, 32, 1024, 1.0, 1.0, GenTaps, 1234ULL},
  {0.011, 8, 1024, 1.0, 1.0, GenTaps, 1234ULL},
  {0.0055, 32, 1024, 1.0, 1.0, GenKiss99, 1234ULL},
  {0.011, 8, 1024, 1.0, 1.0, GenKiss99, 1234ULL}};
TEST_P(RngNormalTableTestD, Result) {
  double meanvar[2];
  getExpectedMeanVar(meanvar);
  ASSERT_TRUE(match(meanvar[0], h_stats[0],
                    CompareApprox<double>(num_sigma * params.tolerance)));
  ASSERT_TRUE(match(meanvar[1], h_stats[1],
                    CompareApprox<double>(num_sigma * params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(RngNormalTableTests, RngNormalTableTestD,
                        ::testing::ValuesIn(inputsd_t));

}  // end namespace Random
}  // end namespace MLCommon
