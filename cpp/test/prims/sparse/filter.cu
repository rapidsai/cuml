/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <raft/random/rng.cuh>

#include <sparse/op/sort.h>
#include <raft/mr/device/allocator.hpp>
#include <sparse/coo.cuh>
#include <sparse/op/filter.cuh>

#include <iostream>

namespace raft {
namespace sparse {

template <typename T>
struct SparseFilterInputs {
  int m, n, nnz;
  unsigned long long int seed;
};

template <typename T>
class SparseFilterTests
  : public ::testing::TestWithParam<SparseFilterInputs<T>> {
 protected:
  void SetUp() override {}

  void TearDown() override {}

 protected:
  SparseFilterInputs<T> params;
};

const std::vector<SparseFilterInputs<float>> inputsf = {{5, 10, 5, 1234ULL}};

typedef SparseFilterTests<float> COORemoveZeros;
TEST_P(COORemoveZeros, Result) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  std::shared_ptr<raft::mr::device::allocator> alloc(
    new raft::mr::device::default_allocator);
  params = ::testing::TestWithParam<SparseFilterInputs<float>>::GetParam();

  float *in_h_vals = new float[params.nnz];

  COO<float> in(alloc, stream, params.nnz, 5, 5);

  raft::random::Rng r(params.seed);
  r.uniform(in.vals(), params.nnz, float(-1.0), float(1.0), stream);

  raft::update_host(in_h_vals, in.vals(), params.nnz, stream);

  in_h_vals[0] = 0;
  in_h_vals[2] = 0;
  in_h_vals[3] = 0;

  int *in_h_rows = new int[params.nnz];
  int *in_h_cols = new int[params.nnz];

  for (int i = 0; i < params.nnz; i++) {
    in_h_rows[i] = params.nnz - i - 1;
    in_h_cols[i] = i;
  }

  raft::update_device(in.rows(), in_h_rows, params.nnz, stream);
  raft::update_device(in.cols(), in_h_cols, params.nnz, stream);
  raft::update_device(in.vals(), in_h_vals, params.nnz, stream);

  op::coo_sort<float>(&in, alloc, stream);

  int out_rows_ref_h[2] = {0, 3};
  int out_cols_ref_h[2] = {4, 1};

  float *out_vals_ref_h = (float *)malloc(2 * sizeof(float));
  out_vals_ref_h[0] = in_h_vals[4];
  out_vals_ref_h[1] = in_h_vals[1];

  COO<float> out_ref(alloc, stream, 2, 5, 5);
  COO<float> out(alloc, stream);

  raft::update_device(out_ref.rows(), *&out_rows_ref_h, 2, stream);
  raft::update_device(out_ref.cols(), *&out_cols_ref_h, 2, stream);
  raft::update_device(out_ref.vals(), out_vals_ref_h, 2, stream);

  op::coo_remove_zeros<32, float>(&in, &out, alloc, stream);

  ASSERT_TRUE(raft::devArrMatch<int>(out_ref.rows(), out.rows(), 2,
                                     raft::Compare<int>()));
  ASSERT_TRUE(raft::devArrMatch<int>(out_ref.cols(), out.cols(), 2,
                                     raft::Compare<int>()));
  ASSERT_TRUE(raft::devArrMatch<float>(out_ref.vals(), out.vals(), 2,
                                       raft::Compare<float>()));

  CUDA_CHECK(hipStreamDestroy(stream));
  free(out_vals_ref_h);

  delete[] in_h_rows;
  delete[] in_h_cols;
  delete[] in_h_vals;
}

INSTANTIATE_TEST_CASE_P(SparseFilterTests, COORemoveZeros,
                        ::testing::ValuesIn(inputsf));

}  // namespace sparse
}  // namespace raft
