/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipsparse.h>
#include <raft/cudart_utils.h>
#include <common/device_buffer.hpp>

#include <gtest/gtest.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <test_utils.h>
#include <sparse/csr.cuh>

namespace MLCommon {
namespace Sparse {

using namespace raft;

template <typename value_idx, typename value_t>
struct CSRToDenseInputs {
  value_idx nrows;
  value_idx ncols;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_t> out_ref_h;
};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const CSRToDenseInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class CSRToDenseTest
  : public ::testing::TestWithParam<CSRToDenseInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.indptr_h;
    std::vector<value_idx> indices_h = params.indices_h;
    std::vector<value_t> data_h = params.data_h;

    allocate(indptr, indptr_h.size());
    allocate(indices, indices_h.size());
    allocate(data, data_h.size());

    update_device(indptr, indptr_h.data(), indptr_h.size(), stream);
    update_device(indices, indices_h.data(), indices_h.size(), stream);
    update_device(data, data_h.data(), data_h.size(), stream);

    std::vector<value_t> out_ref_h = params.out_ref_h;

    allocate(out_ref, out_ref_h.size());

    update_device(out_ref, out_ref_h.data(), out_ref_h.size(), stream);

    allocate(out, out_ref_h.size());
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      CSRToDenseInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    make_data();

    ML::Logger::get().setLevel(CUML_LEVEL_INFO);

    csr_to_dense(handle, params.nrows, params.ncols, indptr, indices, data,
                 params.nrows, out, stream, true);

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUSPARSE_CHECK(hipsparseDestroy(handle));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out));
    CUDA_CHECK(hipFree(out_ref));
  }

  void compare() {
    ASSERT_TRUE(
      devArrMatch(out, out_ref, params.out_ref_h.size(), Compare<value_t>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t handle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_t *out;

  // expected output data
  value_t *out_ref;

  CSRToDenseInputs<value_idx, value_t> params;
};

const std::vector<CSRToDenseInputs<int, float>> inputs_i32_f = {
  {4,
   4,
   {0, 2, 4, 6, 8},
   {0, 1, 2, 3, 0, 1, 2, 3},  // indices
   {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
   {1.0f, 3.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.0f, 5.0f, 50.0f, 28.0f, 0.0f, 0.0f,
    0.0f, 0.0f, 16.0f, 2.0f}},
};
typedef CSRToDenseTest<int, float> CSRToDenseTestF;
TEST_P(CSRToDenseTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(CSRToDenseTest, CSRToDenseTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Sparse
};  // end namespace MLCommon
