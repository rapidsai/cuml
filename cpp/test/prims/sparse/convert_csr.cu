#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <raft/random/rng.cuh>

#include <raft/mr/device/allocator.hpp>
#include <sparse/convert/csr.cuh>
#include <sparse/coo.cuh>

#include <iostream>

namespace raft {
namespace sparse {

/**************************** sorted COO to CSR ****************************/

template <typename T>
struct SparseConvertCSRInputs {
  int m, n, nnz;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os,
                           const SparseConvertCSRInputs<T> &dims) {
  return os;
}

template <typename T>
class SparseConvertCSRTest
  : public ::testing::TestWithParam<SparseConvertCSRInputs<T>> {
 protected:
  void SetUp() override {}

  void TearDown() override {}

 protected:
  SparseConvertCSRInputs<T> params;
};

const std::vector<SparseConvertCSRInputs<float>> inputsf = {
  {5, 10, 5, 1234ULL}};

typedef SparseConvertCSRTest<float> SortedCOOToCSR;
TEST_P(SortedCOOToCSR, Result) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  std::shared_ptr<raft::mr::device::allocator> alloc(
    new raft::mr::device::default_allocator);

  int nnz = 8;

  int *in, *out, *exp;

  int *in_h = new int[nnz]{0, 0, 1, 1, 2, 2, 3, 3};
  int *exp_h = new int[4]{0, 2, 4, 6};

  raft::allocate(in, nnz, true);
  raft::allocate(exp, 4, true);
  raft::allocate(out, 4, true);

  raft::update_device(in, in_h, nnz, stream);
  raft::update_device(exp, exp_h, 4, stream);

  convert::sorted_coo_to_csr<int>(in, nnz, out, 4, alloc, stream);

  ASSERT_TRUE(raft::devArrMatch<int>(out, exp, 4, raft::Compare<int>()));

  hipStreamDestroy(stream);

  delete[] in_h;
  delete[] exp_h;

  CUDA_CHECK(hipFree(in));
  CUDA_CHECK(hipFree(exp));
  CUDA_CHECK(hipFree(out));
}

INSTANTIATE_TEST_CASE_P(SparseConvertCSRTest, SortedCOOToCSR,
                        ::testing::ValuesIn(inputsf));

/******************************** adj graph ********************************/

template <typename Index_>
struct CSRAdjGraphInputs {
  Index_ n_rows;
  Index_ n_cols;
  std::vector<Index_> row_ind;
  std::vector<uint8_t> adj;  // To avoid vector<bool> optimization
  std::vector<Index_> verify;
};

template <typename Index_>
class CSRAdjGraphTest
  : public ::testing::TestWithParam<CSRAdjGraphInputs<Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<CSRAdjGraphInputs<Index_>>::GetParam();
    hipStreamCreate(&stream);
    nnz = params.verify.size();

    raft::allocate(row_ind, params.n_rows);
    raft::allocate(adj, params.n_rows * params.n_cols);
    raft::allocate(result, nnz, true);
    raft::allocate(verify, nnz);
  }

  void Run() {
    raft::update_device(row_ind, params.row_ind.data(), params.n_rows, stream);
    raft::update_device(adj, reinterpret_cast<bool *>(params.adj.data()),
                        params.n_rows * params.n_cols, stream);
    raft::update_device(verify, params.verify.data(), nnz, stream);

    convert::csr_adj_graph_batched<Index_, 32>(
      row_ind, params.n_cols, nnz, params.n_rows, adj, result, stream);

    ASSERT_TRUE(
      raft::devArrMatch<Index_>(verify, result, nnz, raft::Compare<Index_>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(row_ind));
    CUDA_CHECK(hipFree(adj));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    hipStreamDestroy(stream);
  }

 protected:
  CSRAdjGraphInputs<Index_> params;
  hipStream_t stream;
  Index_ nnz;
  Index_ *row_ind, *result, *verify;
  bool *adj;
};

using CSRAdjGraphTestI = CSRAdjGraphTest<int>;
TEST_P(CSRAdjGraphTestI, Result) { Run(); }

using CSRAdjGraphTestL = CSRAdjGraphTest<int64_t>;
TEST_P(CSRAdjGraphTestL, Result) { Run(); }

const std::vector<CSRAdjGraphInputs<int>> csradjgraph_inputs_i = {
  {3,
   6,
   {0, 3, 6},
   {1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
   {0, 1, 2, 0, 1, 2, 0, 1, 2}},
};
const std::vector<CSRAdjGraphInputs<int64_t>> csradjgraph_inputs_l = {
  {3,
   6,
   {0, 3, 6},
   {1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
   {0, 1, 2, 0, 1, 2, 0, 1, 2}},
};

INSTANTIATE_TEST_CASE_P(SparseConvertCSRTest, CSRAdjGraphTestI,
                        ::testing::ValuesIn(csradjgraph_inputs_i));
INSTANTIATE_TEST_CASE_P(SparseConvertCSRTest, CSRAdjGraphTestL,
                        ::testing::ValuesIn(csradjgraph_inputs_l));

}  // namespace sparse
}  // namespace raft
