/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <raft/random/rng.cuh>

#include <sparse/op/sort.h>
#include <raft/mr/device/allocator.hpp>

#include <iostream>

namespace raft {
namespace sparse {

template <typename T>
struct SparseSortInput {
  int m, n, nnz;
  unsigned long long int seed;
};

template <typename T>
class SparseSortTest : public ::testing::TestWithParam<SparseSortInput<T>> {
 protected:
  void SetUp() override {}

  void TearDown() override {}

 protected:
  SparseSortInput<T> params;
};

const std::vector<SparseSortInput<float>> inputsf = {{5, 10, 5, 1234ULL}};

typedef SparseSortTest<float> COOSort;
TEST_P(COOSort, Result) {
  int *in_rows, *in_cols, *verify;
  float *in_vals;

  params = ::testing::TestWithParam<SparseSortInput<float>>::GetParam();
  raft::random::Rng r(params.seed);
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  std::shared_ptr<raft::mr::device::allocator> alloc(
    new raft::mr::device::default_allocator);

  raft::allocate(in_vals, params.nnz);
  r.uniform(in_vals, params.nnz, float(-1.0), float(1.0), stream);

  int *in_rows_h = (int *)malloc(params.nnz * sizeof(int));
  int *in_cols_h = (int *)malloc(params.nnz * sizeof(int));
  int *verify_h = (int *)malloc(params.nnz * sizeof(int));

  for (int i = 0; i < params.nnz; i++) {
    in_rows_h[i] = params.nnz - i - 1;
    verify_h[i] = i;
    in_cols_h[i] = i;
  }

  raft::allocate(in_rows, params.nnz);
  raft::allocate(in_cols, params.nnz);
  raft::allocate(verify, params.nnz);

  raft::update_device(in_rows, in_rows_h, params.nnz, stream);

  raft::update_device(in_cols, in_cols_h, params.nnz, stream);
  raft::update_device(verify, verify_h, params.nnz, stream);

  op::coo_sort(params.m, params.n, params.nnz, in_rows, in_cols, in_vals, alloc,
               stream);

  ASSERT_TRUE(
    raft::devArrMatch<int>(verify, in_rows, params.nnz, raft::Compare<int>()));

  delete[] in_rows_h;
  delete[] in_cols_h;
  delete[] verify_h;

  CUDA_CHECK(hipFree(in_rows));
  CUDA_CHECK(hipFree(in_cols));
  CUDA_CHECK(hipFree(in_vals));
  CUDA_CHECK(hipFree(verify));
  CUDA_CHECK(hipStreamDestroy(stream));
}

INSTANTIATE_TEST_CASE_P(SparseSortTest, COOSort, ::testing::ValuesIn(inputsf));

}  // namespace sparse
}  // namespace raft
