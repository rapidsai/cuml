/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipsparse.h>
#include <raft/cudart_utils.h>
#include <common/device_buffer.hpp>

#include <gtest/gtest.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <test_utils.h>
#include <sparse/csr.cuh>

namespace MLCommon {
namespace Sparse {

using namespace raft;

template <typename value_idx, typename value_t>
struct CSRTransposeInputs {
  value_idx nrows;
  value_idx ncols;
  value_idx nnz;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_idx> out_indptr_ref_h;
  std::vector<value_idx> out_indices_ref_h;
  std::vector<value_t> out_data_ref_h;
};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const CSRTransposeInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class CSRTransposeTest
  : public ::testing::TestWithParam<CSRTransposeInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.indptr_h;
    std::vector<value_idx> indices_h = params.indices_h;
    std::vector<value_t> data_h = params.data_h;

    allocate(indptr, indptr_h.size());
    allocate(indices, indices_h.size());
    allocate(data, data_h.size());

    update_device(indptr, indptr_h.data(), indptr_h.size(), stream);
    update_device(indices, indices_h.data(), indices_h.size(), stream);
    update_device(data, data_h.data(), data_h.size(), stream);

    std::vector<value_idx> out_indptr_ref_h = params.out_indptr_ref_h;
    std::vector<value_idx> out_indices_ref_h = params.out_indices_ref_h;
    std::vector<value_t> out_data_ref_h = params.out_data_ref_h;

    allocate(out_indptr_ref, out_indptr_ref_h.size());
    allocate(out_indices_ref, out_indices_ref_h.size());
    allocate(out_data_ref, out_data_ref_h.size());

    update_device(out_indptr_ref, out_indptr_ref_h.data(),
                  out_indptr_ref_h.size(), stream);
    update_device(out_indices_ref, out_indices_ref_h.data(),
                  out_indices_ref_h.size(), stream);
    update_device(out_data_ref, out_data_ref_h.data(), out_data_ref_h.size(),
                  stream);

    allocate(out_indptr, out_indptr_ref_h.size());
    allocate(out_indices, out_indices_ref_h.size());
    allocate(out_data, out_data_ref_h.size());
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      CSRTransposeInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    make_data();

    ML::Logger::get().setLevel(CUML_LEVEL_INFO);

    csr_transpose(handle, indptr, indices, data, out_indptr, out_indices,
                  out_data, params.nrows, params.ncols, params.nnz, alloc,
                  stream);

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUSPARSE_CHECK(hipsparseDestroy(handle));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_indptr));
    CUDA_CHECK(hipFree(out_indices));
    CUDA_CHECK(hipFree(out_data));
    CUDA_CHECK(hipFree(out_indptr_ref));
    CUDA_CHECK(hipFree(out_indices_ref));
    CUDA_CHECK(hipFree(out_data_ref));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_indptr, out_indptr_ref,
                            params.out_indptr_ref_h.size(),
                            Compare<value_t>()));
    ASSERT_TRUE(devArrMatch(out_indices, out_indices_ref,
                            params.out_indices_ref_h.size(),
                            Compare<value_t>()));
    ASSERT_TRUE(devArrMatch(out_data, out_data_ref,
                            params.out_data_ref_h.size(), Compare<value_t>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t handle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_idx *out_indptr, *out_indices;
  value_t *out_data;

  // expected output data
  value_idx *out_indptr_ref, *out_indices_ref;
  value_t *out_data_ref;

  CSRTransposeInputs<value_idx, value_t> params;
};

const std::vector<CSRTransposeInputs<int, float>> inputs_i32_f = {
  {
    4,
    2,
    8,
    {0, 2, 4, 6, 8},
    {0, 1, 0, 1, 0, 1, 0, 1},  // indices
    {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
    {0, 4, 8},
    {0, 1, 2, 3, 0, 1, 2, 3},  // indices
    {1.0f, 1.0f, 50.0f, 16.0f, 3.0f, 5.0f, 28.0f, 2.0f},
  },
};
typedef CSRTransposeTest<int, float> CSRTransposeTestF;
TEST_P(CSRTransposeTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(CSRTransposeTest, CSRTransposeTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Sparse
};  // end namespace MLCommon
