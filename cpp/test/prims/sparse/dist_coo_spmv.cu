#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include <hipsparse.h>

#include <raft/cudart_utils.h>
#include <raft/linalg/distance_type.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <raft/linalg/unary_op.cuh>
#include <raft/mr/device/allocator.hpp>

#include <sparse/convert/coo.cuh>
#include <sparse/distance/coo_spmv.cuh>
#include <sparse/distance/operators.cuh>

#include <test_utils.h>

namespace raft {
namespace sparse {
namespace distance {

using namespace raft;
using namespace raft::sparse;

template <typename value_idx, typename value_t>
struct SparseDistanceCOOSPMVInputs {
  value_idx n_cols;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_t> out_dists_ref_h;

  raft::distance::DistanceType metric;

  float metric_arg = 0.0;
};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(
  ::std::ostream &os,
  const SparseDistanceCOOSPMVInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class SparseDistanceCOOSPMVTest
  : public ::testing::TestWithParam<
      SparseDistanceCOOSPMVInputs<value_idx, value_t>> {
 public:
  template <typename reduce_f, typename accum_f, typename write_f>
  void compute_dist(reduce_f reduce_func, accum_f accum_func,
                    write_f write_func, bool rev = true) {
    raft::mr::device::buffer<value_idx> coo_rows(
      dist_config.allocator, dist_config.stream,
      max(dist_config.b_nnz, dist_config.a_nnz));

    raft::sparse::convert::csr_to_coo(dist_config.b_indptr, dist_config.b_nrows,
                                      coo_rows.data(), dist_config.b_nnz,
                                      dist_config.stream);

    balanced_coo_pairwise_generalized_spmv<value_idx, value_t>(
      out_dists, dist_config, coo_rows.data(), reduce_func, accum_func,
      write_func);

    if (rev) {
      raft::sparse::convert::csr_to_coo(dist_config.a_indptr,
                                        dist_config.a_nrows, coo_rows.data(),
                                        dist_config.a_nnz, dist_config.stream);

      balanced_coo_pairwise_generalized_spmv_rev<value_idx, value_t>(
        out_dists, dist_config, coo_rows.data(), reduce_func, accum_func,
        write_func);
    }
  }

  void run_spmv() {
    switch (params.metric) {
      case raft::distance::DistanceType::InnerProduct:
        compute_dist(Product(), Sum(), AtomicAdd(), true);
        break;
      case raft::distance::DistanceType::L2Unexpanded:
        compute_dist(SqDiff(), Sum(), AtomicAdd());
        break;
      case raft::distance::DistanceType::Canberra:
        compute_dist(
          [] __device__(value_t a, value_t b) {
            return fabsf(a - b) / (fabsf(a) + fabsf(b));
          },
          Sum(), AtomicAdd());
        break;
      case raft::distance::DistanceType::L1:
        compute_dist(AbsDiff(), Sum(), AtomicAdd());
        break;
      case raft::distance::DistanceType::Linf:
        compute_dist(AbsDiff(), Max(), AtomicMax());
        break;
      case raft::distance::DistanceType::LpUnexpanded: {
        compute_dist(PDiff(params.metric_arg), Sum(), AtomicAdd());
        float p = 1.0f / params.metric_arg;
        raft::linalg::unaryOp<value_t>(
          out_dists, out_dists, dist_config.a_nrows * dist_config.b_nrows,
          [=] __device__(value_t input) { return powf(input, p); },
          dist_config.stream);

      } break;
      default:
        throw raft::exception("Unknown distance");
    }
  }

 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.indptr_h;
    std::vector<value_idx> indices_h = params.indices_h;
    std::vector<value_t> data_h = params.data_h;

    allocate(indptr, indptr_h.size());
    allocate(indices, indices_h.size());
    allocate(data, data_h.size());

    update_device(indptr, indptr_h.data(), indptr_h.size(), stream);
    update_device(indices, indices_h.data(), indices_h.size(), stream);
    update_device(data, data_h.data(), data_h.size(), stream);

    std::vector<value_t> out_dists_ref_h = params.out_dists_ref_h;

    allocate(out_dists_ref, (indptr_h.size() - 1) * (indptr_h.size() - 1));

    update_device(out_dists_ref, out_dists_ref_h.data(), out_dists_ref_h.size(),
                  stream);
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      SparseDistanceCOOSPMVInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<raft::mr::device::allocator> alloc(
      new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    make_data();

    dist_config.b_nrows = params.indptr_h.size() - 1;
    dist_config.b_ncols = params.n_cols;
    dist_config.b_nnz = params.indices_h.size();
    dist_config.b_indptr = indptr;
    dist_config.b_indices = indices;
    dist_config.b_data = data;
    dist_config.a_nrows = params.indptr_h.size() - 1;
    dist_config.a_ncols = params.n_cols;
    dist_config.a_nnz = params.indices_h.size();
    dist_config.a_indptr = indptr;
    dist_config.a_indices = indices;
    dist_config.a_data = data;
    dist_config.handle = cusparseHandle;
    dist_config.allocator = alloc;
    dist_config.stream = stream;

    int out_size = dist_config.a_nrows * dist_config.b_nrows;

    allocate(out_dists, out_size);

    ML::Logger::get().setLevel(CUML_LEVEL_DEBUG);

    run_spmv();

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_dists));
    CUDA_CHECK(hipFree(out_dists_ref));
  }

  void compare() {
    raft::print_device_vector("expected: ", out_dists_ref,
                              params.out_dists_ref_h.size(), std::cout);
    raft::print_device_vector("out_dists: ", out_dists,
                              params.out_dists_ref_h.size(), std::cout);
    ASSERT_TRUE(devArrMatch(out_dists_ref, out_dists,
                            params.out_dists_ref_h.size(),
                            CompareApprox<value_t>(1e-3)));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t cusparseHandle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_t *out_dists, *out_dists_ref;

  raft::sparse::distance::distances_config_t<value_idx, value_t> dist_config;

  SparseDistanceCOOSPMVInputs<value_idx, value_t> params;
};

const std::vector<SparseDistanceCOOSPMVInputs<int, float>> inputs_i32_f = {
  {2,
   {0, 2, 4, 6, 8},
   {0, 1, 0, 1, 0, 1, 0, 1},
   {1.0f, 2.0f, 1.0f, 2.0f, 1.0f, 2.0f, 1.0f, 2.0f},
   {5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0,
    5.0},
   raft::distance::DistanceType::InnerProduct},
  {2,
   {0, 2, 4, 6, 8},
   {0, 1, 0, 1, 0, 1, 0, 1},  // indices
   {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
   {
     // dense output
     0.0,
     4.0,
     3026.0,
     226.0,
     4.0,
     0.0,
     2930.0,
     234.0,
     3026.0,
     2930.0,
     0.0,
     1832.0,
     226.0,
     234.0,
     1832.0,
     0.0,
   },
   raft::distance::DistanceType::L2Unexpanded},

  {10,
   {0, 5, 11, 15, 20, 27, 32, 36, 43, 47, 50},
   {0, 1, 3, 6, 8, 0, 1, 2, 3, 5, 6, 1, 2, 4, 8, 0, 2,
    3, 4, 7, 0, 1, 2, 3, 4, 6, 8, 0, 1, 2, 5, 7, 1, 5,
    8, 9, 0, 1, 2, 5, 6, 8, 9, 2, 4, 5, 7, 0, 3, 9},  // indices
   {0.5438, 0.2695, 0.4377, 0.7174, 0.9251, 0.7648, 0.3322, 0.7279, 0.4131,
    0.5167, 0.8655, 0.0730, 0.0291, 0.9036, 0.7988, 0.5019, 0.7663, 0.2190,
    0.8206, 0.3625, 0.0411, 0.3995, 0.5688, 0.7028, 0.8706, 0.3199, 0.4431,
    0.0535, 0.2225, 0.8853, 0.1932, 0.3761, 0.3379, 0.1771, 0.2107, 0.228,
    0.5279, 0.4885, 0.3495, 0.5079, 0.2325, 0.2331, 0.3018, 0.6231, 0.2645,
    0.8429, 0.6625, 0.0797, 0.2724, 0.4218},
   {0.0,
    3.3954660629919076,
    5.6469232737388815,
    6.373112846266441,
    4.0212880272531715,
    6.916281504639404,
    5.741508386786526,
    5.411470999663036,
    9.0,
    4.977014354725805,
    3.3954660629919076,
    0.0,
    7.56256082439209,
    5.540261147481582,
    4.832322929216881,
    4.62003193872216,
    6.498056792320361,
    4.309846252268695,
    6.317531174829905,
    6.016362684141827,
    5.6469232737388815,
    7.56256082439209,
    0.0,
    5.974878731322299,
    4.898357301336036,
    6.442097410320605,
    5.227077347287883,
    7.134101195584642,
    5.457753923371659,
    7.0,
    6.373112846266441,
    5.540261147481582,
    5.974878731322299,
    0.0,
    5.5507273748583,
    4.897749658726415,
    9.0,
    8.398776718824767,
    3.908281400328807,
    4.83431066343688,
    4.0212880272531715,
    4.832322929216881,
    4.898357301336036,
    5.5507273748583,
    0.0,
    6.632989819428174,
    7.438852294822894,
    5.6631570310967465,
    7.579428202635459,
    6.760811985364303,
    6.916281504639404,
    4.62003193872216,
    6.442097410320605,
    4.897749658726415,
    6.632989819428174,
    0.0,
    5.249404187382862,
    6.072559523278559,
    4.07661278488929,
    6.19678948003145,
    5.741508386786526,
    6.498056792320361,
    5.227077347287883,
    9.0,
    7.438852294822894,
    5.249404187382862,
    0.0,
    3.854811639654704,
    6.652724827169063,
    5.298236851430971,
    5.411470999663036,
    4.309846252268695,
    7.134101195584642,
    8.398776718824767,
    5.6631570310967465,
    6.072559523278559,
    3.854811639654704,
    0.0,
    7.529184598969917,
    6.903282911791188,
    9.0,
    6.317531174829905,
    5.457753923371659,
    3.908281400328807,
    7.579428202635459,
    4.07661278488929,
    6.652724827169063,
    7.529184598969917,
    0.0,
    7.0,
    4.977014354725805,
    6.016362684141827,
    7.0,
    4.83431066343688,
    6.760811985364303,
    6.19678948003145,
    5.298236851430971,
    6.903282911791188,
    7.0,
    0.0},
   raft::distance::DistanceType::Canberra},

  {10,
   {0, 5, 11, 15, 20, 27, 32, 36, 43, 47, 50},
   {0, 1, 3, 6, 8, 0, 1, 2, 3, 5, 6, 1, 2, 4, 8, 0, 2,
    3, 4, 7, 0, 1, 2, 3, 4, 6, 8, 0, 1, 2, 5, 7, 1, 5,
    8, 9, 0, 1, 2, 5, 6, 8, 9, 2, 4, 5, 7, 0, 3, 9},  // indices
   {0.5438, 0.2695, 0.4377, 0.7174, 0.9251, 0.7648, 0.3322, 0.7279, 0.4131,
    0.5167, 0.8655, 0.0730, 0.0291, 0.9036, 0.7988, 0.5019, 0.7663, 0.2190,
    0.8206, 0.3625, 0.0411, 0.3995, 0.5688, 0.7028, 0.8706, 0.3199, 0.4431,
    0.0535, 0.2225, 0.8853, 0.1932, 0.3761, 0.3379, 0.1771, 0.2107, 0.228,
    0.5279, 0.4885, 0.3495, 0.5079, 0.2325, 0.2331, 0.3018, 0.6231, 0.2645,
    0.8429, 0.6625, 0.0797, 0.2724, 0.4218},
   {0.0,
    1.31462855332296,
    1.3690307816129905,
    1.698603990921237,
    1.3460470789553531,
    1.6636670712582544,
    1.2651744044972217,
    1.1938329352055201,
    1.8811409082590185,
    1.3653115050624267,
    1.31462855332296,
    0.0,
    1.9447722703291133,
    1.42818777206562,
    1.4685491458946494,
    1.3071999866010466,
    1.4988622861692171,
    0.9698559287406783,
    1.4972023224597841,
    1.5243383567266802,
    1.3690307816129905,
    1.9447722703291133,
    0.0,
    1.2748400840107568,
    1.0599569946448246,
    1.546591282841402,
    1.147526531928459,
    1.447002179128145,
    1.5982242387673176,
    1.3112533607072414,
    1.698603990921237,
    1.42818777206562,
    1.2748400840107568,
    0.0,
    1.038121552545461,
    1.011788365364402,
    1.3907391109256988,
    1.3128200942311496,
    1.19595706584447,
    1.3233328139624725,
    1.3460470789553531,
    1.4685491458946494,
    1.0599569946448246,
    1.038121552545461,
    0.0,
    1.3642741698145529,
    1.3493868683808095,
    1.394942694628328,
    1.572881849642552,
    1.380122665319464,
    1.6636670712582544,
    1.3071999866010466,
    1.546591282841402,
    1.011788365364402,
    1.3642741698145529,
    0.0,
    1.018961640373018,
    1.0114394258945634,
    0.8338711034820684,
    1.1247823842299223,
    1.2651744044972217,
    1.4988622861692171,
    1.147526531928459,
    1.3907391109256988,
    1.3493868683808095,
    1.018961640373018,
    0.0,
    0.7701238110357329,
    1.245486437864406,
    0.5551259549534626,
    1.1938329352055201,
    0.9698559287406783,
    1.447002179128145,
    1.3128200942311496,
    1.394942694628328,
    1.0114394258945634,
    0.7701238110357329,
    0.0,
    1.1886800117391216,
    1.0083692448135637,
    1.8811409082590185,
    1.4972023224597841,
    1.5982242387673176,
    1.19595706584447,
    1.572881849642552,
    0.8338711034820684,
    1.245486437864406,
    1.1886800117391216,
    0.0,
    1.3661374102525012,
    1.3653115050624267,
    1.5243383567266802,
    1.3112533607072414,
    1.3233328139624725,
    1.380122665319464,
    1.1247823842299223,
    0.5551259549534626,
    1.0083692448135637,
    1.3661374102525012,
    0.0},
   raft::distance::DistanceType::LpUnexpanded,
   2.0},

  {10,
   {0, 5, 11, 15, 20, 27, 32, 36, 43, 47, 50},
   {0, 1, 3, 6, 8, 0, 1, 2, 3, 5, 6, 1, 2, 4, 8, 0, 2,
    3, 4, 7, 0, 1, 2, 3, 4, 6, 8, 0, 1, 2, 5, 7, 1, 5,
    8, 9, 0, 1, 2, 5, 6, 8, 9, 2, 4, 5, 7, 0, 3, 9},  // indices
   {0.5438, 0.2695, 0.4377, 0.7174, 0.9251, 0.7648, 0.3322, 0.7279, 0.4131,
    0.5167, 0.8655, 0.0730, 0.0291, 0.9036, 0.7988, 0.5019, 0.7663, 0.2190,
    0.8206, 0.3625, 0.0411, 0.3995, 0.5688, 0.7028, 0.8706, 0.3199, 0.4431,
    0.0535, 0.2225, 0.8853, 0.1932, 0.3761, 0.3379, 0.1771, 0.2107, 0.228,
    0.5279, 0.4885, 0.3495, 0.5079, 0.2325, 0.2331, 0.3018, 0.6231, 0.2645,
    0.8429, 0.6625, 0.0797, 0.2724, 0.4218},
   {0.0,
    0.9251771844789913,
    0.9036452083899731,
    0.9251771844789913,
    0.8706483735804971,
    0.9251771844789913,
    0.717493881903289,
    0.6920214832303888,
    0.9251771844789913,
    0.9251771844789913,
    0.9251771844789913,
    0.0,
    0.9036452083899731,
    0.8655339692155823,
    0.8706483735804971,
    0.8655339692155823,
    0.8655339692155823,
    0.6329837991017668,
    0.8655339692155823,
    0.8655339692155823,
    0.9036452083899731,
    0.9036452083899731,
    0.0,
    0.7988276152181608,
    0.7028075145996631,
    0.9036452083899731,
    0.9036452083899731,
    0.9036452083899731,
    0.8429599432532096,
    0.9036452083899731,
    0.9251771844789913,
    0.8655339692155823,
    0.7988276152181608,
    0.0,
    0.48376552205293305,
    0.8206394616536681,
    0.8206394616536681,
    0.8206394616536681,
    0.8429599432532096,
    0.8206394616536681,
    0.8706483735804971,
    0.8706483735804971,
    0.7028075145996631,
    0.48376552205293305,
    0.0,
    0.8706483735804971,
    0.8706483735804971,
    0.8706483735804971,
    0.8429599432532096,
    0.8706483735804971,
    0.9251771844789913,
    0.8655339692155823,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.0,
    0.8853924473642432,
    0.535821510936138,
    0.6497196601457607,
    0.8853924473642432,
    0.717493881903289,
    0.8655339692155823,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.8853924473642432,
    0.0,
    0.5279604218147174,
    0.6658348373853169,
    0.33799874888632914,
    0.6920214832303888,
    0.6329837991017668,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.535821510936138,
    0.5279604218147174,
    0.0,
    0.662579808115858,
    0.5079750812968089,
    0.9251771844789913,
    0.8655339692155823,
    0.8429599432532096,
    0.8429599432532096,
    0.8429599432532096,
    0.6497196601457607,
    0.6658348373853169,
    0.662579808115858,
    0.0,
    0.8429599432532096,
    0.9251771844789913,
    0.8655339692155823,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.8853924473642432,
    0.33799874888632914,
    0.5079750812968089,
    0.8429599432532096,
    0.0},
   raft::distance::DistanceType::Linf},

  {4,
   {0, 1, 1, 2, 4},
   {3, 2, 0, 1},  // indices
   {0.99296, 0.42180, 0.11687, 0.305869},
   {
     // dense output
     0.0,
     0.99296,
     1.41476,
     1.415707,
     0.99296,
     0.0,
     0.42180,
     0.42274,
     1.41476,
     0.42180,
     0.0,
     0.84454,
     1.41570,
     0.42274,
     0.84454,
     0.0,
   },
   raft::distance::DistanceType::L1}

};

typedef SparseDistanceCOOSPMVTest<int, float> SparseDistanceCOOSPMVTestF;
TEST_P(SparseDistanceCOOSPMVTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(SparseDistanceCOOSPMVTests, SparseDistanceCOOSPMVTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // namespace distance
};  // end namespace sparse
};  // end namespace raft
