/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipsparse.h>
#include <raft/cudart_utils.h>
#include <common/device_buffer.hpp>

#include <gtest/gtest.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <test_utils.h>
#include <sparse/csr.cuh>

namespace MLCommon {
namespace Sparse {

using namespace raft;

template <typename value_idx, typename value_t>
struct CSRRowSliceInputs {
  value_idx start_row;
  value_idx stop_row;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_idx> out_indptr_ref_h;
  std::vector<value_idx> out_indices_ref_h;
  std::vector<value_t> out_data_ref_h;
};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const CSRRowSliceInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class CSRRowSliceTest
  : public ::testing::TestWithParam<CSRRowSliceInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.indptr_h;
    std::vector<value_idx> indices_h = params.indices_h;
    std::vector<value_t> data_h = params.data_h;

    allocate(indptr, indptr_h.size());
    allocate(indices, indices_h.size());
    allocate(data, data_h.size());

    update_device(indptr, indptr_h.data(), indptr_h.size(), stream);
    update_device(indices, indices_h.data(), indices_h.size(), stream);
    update_device(data, data_h.data(), data_h.size(), stream);

    std::vector<value_idx> out_indptr_ref_h = params.out_indptr_ref_h;
    std::vector<value_idx> out_indices_ref_h = params.out_indices_ref_h;
    std::vector<value_t> out_data_ref_h = params.out_data_ref_h;

    allocate(out_indptr_ref, out_indptr_ref_h.size());
    allocate(out_indices_ref, out_indices_ref_h.size());
    allocate(out_data_ref, out_data_ref_h.size());

    update_device(out_indptr_ref, out_indptr_ref_h.data(),
                  out_indptr_ref_h.size(), stream);
    update_device(out_indices_ref, out_indices_ref_h.data(),
                  out_indices_ref_h.size(), stream);
    update_device(out_data_ref, out_data_ref_h.data(), out_data_ref_h.size(),
                  stream);

    allocate(out_indptr, out_indptr_ref_h.size());
    allocate(out_indices, out_indices_ref_h.size());
    allocate(out_data, out_data_ref_h.size());
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      CSRRowSliceInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    make_data();

    ML::Logger::get().setLevel(CUML_LEVEL_INFO);

    int csr_start_offset;
    int csr_stop_offset;

    MLCommon::Sparse::csr_row_slice_indptr(
      params.start_row, params.stop_row, indptr, out_indptr, &csr_start_offset,
      &csr_stop_offset, stream);

    MLCommon::Sparse::csr_row_slice_populate(csr_start_offset, csr_stop_offset,
                                             indices, data, out_indices,
                                             out_data, stream);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_indptr));
    CUDA_CHECK(hipFree(out_indices));
    CUDA_CHECK(hipFree(out_data));
    CUDA_CHECK(hipFree(out_indptr_ref));
    CUDA_CHECK(hipFree(out_indices_ref));
    CUDA_CHECK(hipFree(out_data_ref));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_indptr, out_indptr_ref,
                            params.out_indptr_ref_h.size(),
                            Compare<value_t>()));
    ASSERT_TRUE(devArrMatch(out_indices, out_indices_ref,
                            params.out_indices_ref_h.size(),
                            Compare<value_t>()));
    ASSERT_TRUE(devArrMatch(out_data, out_data_ref,
                            params.out_data_ref_h.size(), Compare<value_t>()));
  }

 protected:
  hipStream_t stream;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_idx *out_indptr, *out_indices;
  value_t *out_data;

  // expected output data
  value_idx *out_indptr_ref, *out_indices_ref;
  value_t *out_data_ref;

  CSRRowSliceInputs<value_idx, value_t> params;
};

const std::vector<CSRRowSliceInputs<int, float>> inputs_i32_f = {
  {1,
   3,
   {0, 2, 4, 6, 8},
   {0, 1, 0, 1, 0, 1, 0, 1},  // indices
   {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
   {0, 2, 4, 6},
   {0, 1, 0, 1, 0, 1},  // indices
   {1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f}},
  {
    2,
    3,
    {0, 2, 4, 6, 8},
    {0, 1, 0, 1, 0, 1, 0, 1},  // indices
    {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
    {0, 2, 4},
    {0, 1, 0, 1},  // indices
    {50.0f, 28.0f, 16.0f, 2.0f},
  }

};
typedef CSRRowSliceTest<int, float> CSRRowSliceTestF;
TEST_P(CSRRowSliceTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(CSRRowSliceTest, CSRRowSliceTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Sparse
};  // end namespace MLCommon
