/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include <sparse/csr.cuh>
#include <sparse/linalg/add.cuh>

#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <raft/random/rng.cuh>

#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

template <typename Type_f, typename Index_>
struct CSRMatrixVal {
  std::vector<Index_> row_ind;
  std::vector<Index_> row_ind_ptr;
  std::vector<Type_f> values;
};

template <typename Type_f, typename Index_>
struct CSRAddInputs {
  CSRMatrixVal<Type_f, Index_> matrix_a;
  CSRMatrixVal<Type_f, Index_> matrix_b;
  CSRMatrixVal<Type_f, Index_> matrix_verify;
};

template <typename Type_f, typename Index_>
class CSRAddTest
  : public ::testing::TestWithParam<CSRAddInputs<Type_f, Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<CSRAddInputs<Type_f, Index_>>::GetParam();
    n_rows = params.matrix_a.row_ind.size();
    nnz_a = params.matrix_a.row_ind_ptr.size();
    nnz_b = params.matrix_b.row_ind_ptr.size();
    nnz_result = params.matrix_verify.row_ind_ptr.size();

    hipStreamCreate(&stream);

    raft::allocate(ind_a, n_rows);
    raft::allocate(ind_ptr_a, nnz_a);
    raft::allocate(values_a, nnz_a);

    raft::allocate(ind_b, n_rows);
    raft::allocate(ind_ptr_b, nnz_b);
    raft::allocate(values_b, nnz_b);

    raft::allocate(ind_verify, n_rows);
    raft::allocate(ind_ptr_verify, nnz_result);
    raft::allocate(values_verify, nnz_result);

    raft::allocate(ind_result, n_rows);
    raft::allocate(ind_ptr_result, nnz_result);
    raft::allocate(values_result, nnz_result);
  }

  void Run() {
    std::shared_ptr<MLCommon::deviceAllocator> alloc(
      new raft::mr::device::default_allocator);

    raft::update_device(ind_a, params.matrix_a.row_ind.data(), n_rows, stream);
    raft::update_device(ind_ptr_a, params.matrix_a.row_ind_ptr.data(), nnz_a,
                        stream);
    raft::update_device(values_a, params.matrix_a.values.data(), nnz_a, stream);

    raft::update_device(ind_b, params.matrix_b.row_ind.data(), n_rows, stream);
    raft::update_device(ind_ptr_b, params.matrix_b.row_ind_ptr.data(), nnz_b,
                        stream);
    raft::update_device(values_b, params.matrix_b.values.data(), nnz_b, stream);

    raft::update_device(ind_verify, params.matrix_verify.row_ind.data(), n_rows,
                        stream);
    raft::update_device(ind_ptr_verify, params.matrix_verify.row_ind_ptr.data(),
                        nnz_result, stream);
    raft::update_device(values_verify, params.matrix_verify.values.data(),
                        nnz_result, stream);

    Index_ nnz = linalg::csr_add_calc_inds<Type_f, 32>(
      ind_a, ind_ptr_a, values_a, nnz_a, ind_b, ind_ptr_b, values_b, nnz_b,
      n_rows, ind_result, alloc, stream);

    ASSERT_TRUE(nnz == nnz_result);
    ASSERT_TRUE(raft::devArrMatch<Index_>(ind_verify, ind_result, n_rows,
                                          raft::Compare<Index_>()));

    linalg::csr_add_finalize<Type_f, 32>(
      ind_a, ind_ptr_a, values_a, nnz_a, ind_b, ind_ptr_b, values_b, nnz_b,
      n_rows, ind_result, ind_ptr_result, values_result, stream);

    ASSERT_TRUE(raft::devArrMatch<Index_>(ind_ptr_verify, ind_ptr_result, nnz,
                                          raft::Compare<Index_>()));
    ASSERT_TRUE(raft::devArrMatch<Type_f>(values_verify, values_result, nnz,
                                          raft::Compare<Type_f>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(ind_a));
    CUDA_CHECK(hipFree(ind_b));
    CUDA_CHECK(hipFree(ind_result));
    CUDA_CHECK(hipFree(ind_ptr_a));
    CUDA_CHECK(hipFree(ind_ptr_b));
    CUDA_CHECK(hipFree(ind_ptr_verify));
    CUDA_CHECK(hipFree(ind_ptr_result));
    CUDA_CHECK(hipFree(values_a));
    CUDA_CHECK(hipFree(values_b));
    CUDA_CHECK(hipFree(values_verify));
    CUDA_CHECK(hipFree(values_result));
    hipStreamDestroy(stream);
  }

 protected:
  CSRAddInputs<Type_f, Index_> params;
  hipStream_t stream;
  Index_ n_rows, nnz_a, nnz_b, nnz_result;
  Index_ *ind_a, *ind_b, *ind_verify, *ind_result, *ind_ptr_a, *ind_ptr_b,
    *ind_ptr_verify, *ind_ptr_result;
  Type_f *values_a, *values_b, *values_verify, *values_result;
};

using CSRAddTestF = CSRAddTest<float, int>;
TEST_P(CSRAddTestF, Result) { Run(); }

using CSRAddTestD = CSRAddTest<double, int>;
TEST_P(CSRAddTestD, Result) { Run(); }

const std::vector<CSRAddInputs<float, int>> csradd_inputs_f = {
  {{{0, 4, 8, 9},
    {1, 2, 3, 4, 1, 2, 3, 5, 0, 1},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 4, 8, 9},
    {1, 2, 5, 4, 0, 2, 3, 5, 1, 0},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 5, 10, 12},
    {1, 2, 3, 4, 5, 1, 2, 3, 5, 0, 0, 1, 1, 0},
    {2.0, 2.0, 0.5, 1.0, 0.5, 1.0, 2.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0}}},
};
const std::vector<CSRAddInputs<double, int>> csradd_inputs_d = {
  {{{0, 4, 8, 9},
    {1, 2, 3, 4, 1, 2, 3, 5, 0, 1},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 4, 8, 9},
    {1, 2, 5, 4, 0, 2, 3, 5, 1, 0},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 5, 10, 12},
    {1, 2, 3, 4, 5, 1, 2, 3, 5, 0, 0, 1, 1, 0},
    {2.0, 2.0, 0.5, 1.0, 0.5, 1.0, 2.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0}}},
};

INSTANTIATE_TEST_CASE_P(SparseAddTest, CSRAddTestF,
                        ::testing::ValuesIn(csradd_inputs_f));
INSTANTIATE_TEST_CASE_P(SparseAddTest, CSRAddTestD,
                        ::testing::ValuesIn(csradd_inputs_d));

}  // namespace sparse
}  // namespace raft
