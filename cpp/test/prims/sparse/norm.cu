#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <raft/random/rng.cuh>
#include <sparse/csr.cuh>
#include <sparse/linalg/norm.cuh>

#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

enum NormalizeMethod { MAX, L1 };

template <typename Type_f, typename Index_>
struct CSRRowNormalizeInputs {
  NormalizeMethod method;
  std::vector<Index_> ex_scan;
  std::vector<Type_f> in_vals;
  std::vector<Type_f> verify;
};

template <typename Type_f, typename Index_>
class CSRRowNormalizeTest
  : public ::testing::TestWithParam<CSRRowNormalizeInputs<Type_f, Index_>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<
      CSRRowNormalizeInputs<Type_f, Index_>>::GetParam();
    hipStreamCreate(&stream);

    raft::allocate(in_vals, params.in_vals.size());
    raft::allocate(verify, params.verify.size());
    raft::allocate(ex_scan, params.ex_scan.size());
    raft::allocate(result, params.verify.size(), true);
  }

  void Run() {
    Index_ n_rows = params.ex_scan.size();
    Index_ nnz = params.in_vals.size();

    raft::update_device(ex_scan, params.ex_scan.data(), n_rows, stream);
    raft::update_device(in_vals, params.in_vals.data(), nnz, stream);
    raft::update_device(verify, params.verify.data(), nnz, stream);

    switch (params.method) {
      case MAX:
        linalg::csr_row_normalize_max<32, Type_f>(ex_scan, in_vals, nnz, n_rows,
                                                  result, stream);
        break;
      case L1:
        linalg::csr_row_normalize_l1<32, Type_f>(ex_scan, in_vals, nnz, n_rows,
                                                 result, stream);
        break;
    }

    ASSERT_TRUE(
      raft::devArrMatch<Type_f>(verify, result, nnz, raft::Compare<Type_f>()));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(ex_scan));
    CUDA_CHECK(hipFree(in_vals));
    CUDA_CHECK(hipFree(verify));
    CUDA_CHECK(hipFree(result));
    hipStreamDestroy(stream);
  }

 protected:
  CSRRowNormalizeInputs<Type_f, Index_> params;
  hipStream_t stream;
  Index_ *ex_scan;
  Type_f *in_vals, *result, *verify;
};

using CSRRowNormalizeTestF = CSRRowNormalizeTest<float, int>;
TEST_P(CSRRowNormalizeTestF, Result) { Run(); }

using CSRRowNormalizeTestD = CSRRowNormalizeTest<double, int>;
TEST_P(CSRRowNormalizeTestD, Result) { Run(); }

const std::vector<CSRRowNormalizeInputs<float, int>> csrnormalize_inputs_f = {
  {MAX,
   {0, 4, 8, 9},
   {5.0, 1.0, 0.0, 0.0, 10.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {1.0, 0.2, 0.0, 0.0, 1.0, 0.1, 0.0, 0.0, 1, 0.0}},
  {L1,
   {0, 4, 8, 9},
   {1.0, 1.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {0.5, 0.5, 0.0, 0.0, 0.5, 0.5, 0.0, 0.0, 1, 0.0}},
};
const std::vector<CSRRowNormalizeInputs<double, int>> csrnormalize_inputs_d = {
  {MAX,
   {0, 4, 8, 9},
   {5.0, 1.0, 0.0, 0.0, 10.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {1.0, 0.2, 0.0, 0.0, 1.0, 0.1, 0.0, 0.0, 1, 0.0}},
  {L1,
   {0, 4, 8, 9},
   {1.0, 1.0, 0.0, 0.0, 1.0, 1.0, 0.0, 0.0, 1.0, 0.0},
   {0.5, 0.5, 0.0, 0.0, 0.5, 0.5, 0.0, 0.0, 1, 0.0}},
};

INSTANTIATE_TEST_CASE_P(SparseNormTest, CSRRowNormalizeTestF,
                        ::testing::ValuesIn(csrnormalize_inputs_f));
INSTANTIATE_TEST_CASE_P(SparseNormTest, CSRRowNormalizeTestD,
                        ::testing::ValuesIn(csrnormalize_inputs_d));

}  // namespace sparse
}  // namespace raft
