/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipsparse.h>
#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <common/device_buffer.hpp>

#include <raft/sparse/cusparse_wrappers.h>
#include <test_utils.h>
#include <cuml/common/logger.hpp>
#include <sparse/knn.cuh>

namespace MLCommon {
namespace Sparse {
namespace Selection {

using namespace raft;

template <typename value_idx, typename value_t>
struct SparseKNNInputs {
  value_idx n_cols;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_t> out_dists_ref_h;
  std::vector<value_idx> out_indices_ref_h;

  int k;

  int batch_size_index = 2;
  int batch_size_query = 2;

  ML::MetricType metric = ML::MetricType::METRIC_L2;
};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const SparseKNNInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class SparseKNNTest
  : public ::testing::TestWithParam<SparseKNNInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.indptr_h;
    std::vector<value_idx> indices_h = params.indices_h;
    std::vector<value_t> data_h = params.data_h;

    allocate(indptr, indptr_h.size());
    allocate(indices, indices_h.size());
    allocate(data, data_h.size());

    update_device(indptr, indptr_h.data(), indptr_h.size(), stream);
    update_device(indices, indices_h.data(), indices_h.size(), stream);
    update_device(data, data_h.data(), data_h.size(), stream);

    std::vector<value_t> out_dists_ref_h = params.out_dists_ref_h;
    std::vector<value_idx> out_indices_ref_h = params.out_indices_ref_h;

    allocate(out_indices_ref, out_indices_ref_h.size());
    allocate(out_dists_ref, out_dists_ref_h.size());

    update_device(out_indices_ref, out_indices_ref_h.data(),
                  out_indices_ref_h.size(), stream);
    update_device(out_dists_ref, out_dists_ref_h.data(), out_dists_ref_h.size(),
                  stream);

    allocate(out_dists, n_rows * k);
    allocate(out_indices, n_rows * k);
  }

  void SetUp() override {
    params =
      ::testing::TestWithParam<SparseKNNInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    ML::Logger::get().setLevel(CUML_LEVEL_INFO);

    n_rows = params.indptr_h.size() - 1;
    nnz = params.indices_h.size();
    k = params.k;

    make_data();

    brute_force_knn<value_idx, value_t>(
      indptr, indices, data, nnz, n_rows, params.n_cols, indptr, indices, data,
      nnz, n_rows, params.n_cols, out_indices, out_dists, k, cusparseHandle,
      alloc, stream, params.batch_size_index, params.batch_size_query,
      params.metric);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_indices));
    CUDA_CHECK(hipFree(out_dists));
    CUDA_CHECK(hipFree(out_indices_ref));
    CUDA_CHECK(hipFree(out_dists_ref));

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_dists_ref, out_dists, n_rows * k,
                            CompareApprox<value_t>(1e-4)));
    ASSERT_TRUE(devArrMatch(out_indices_ref, out_indices, n_rows * k,
                            Compare<value_idx>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t cusparseHandle;

  int n_rows, nnz, k;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_idx *out_indices;
  value_t *out_dists;

  value_idx *out_indices_ref;
  value_t *out_dists_ref;

  SparseKNNInputs<value_idx, value_t> params;
};

const std::vector<SparseKNNInputs<int, float>> inputs_i32_f = {
  {9,                                                 // ncols
   {0, 2, 4, 6, 8},                                   // indptr
   {0, 4, 0, 3, 0, 2, 0, 8},                          // indices
   {0.0f, 1.0f, 5.0f, 6.0f, 5.0f, 6.0f, 0.0f, 1.0f},  // data
   {0, 1.41421, 0, 7.87401, 0, 7.87401, 0, 1.41421},  // dists
   {0, 3, 1, 0, 2, 0, 3, 0},                          // inds
   2,
   2,
   ML::MetricType::METRIC_L2}};
typedef SparseKNNTest<int, float> KNNTestF;
TEST_P(KNNTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(SparseKNNTest, KNNTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Selection
};  // end namespace Sparse
};  // end namespace MLCommon
