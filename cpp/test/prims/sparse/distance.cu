/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include <hipsparse.h>

#include <raft/cudart_utils.h>
#include <raft/linalg/distance_type.h>
#include <raft/sparse/cusparse_wrappers.h>

#include <common/device_buffer.hpp>

#include <sparse/distance.cuh>

#include <test_utils.h>

namespace MLCommon {
namespace Sparse {
namespace Selection {

using namespace raft;

template <typename value_idx, typename value_t>
struct SparseDistanceInputs {
  value_idx n_cols;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_t> out_dists_ref_h;

  raft::distance::DistanceType metric;
};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(
  ::std::ostream &os, const SparseDistanceInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class SparseDistanceTest
  : public ::testing::TestWithParam<SparseDistanceInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.indptr_h;
    std::vector<value_idx> indices_h = params.indices_h;
    std::vector<value_t> data_h = params.data_h;

    allocate(indptr, indptr_h.size());
    allocate(indices, indices_h.size());
    allocate(data, data_h.size());

    update_device(indptr, indptr_h.data(), indptr_h.size(), stream);
    update_device(indices, indices_h.data(), indices_h.size(), stream);
    update_device(data, data_h.data(), data_h.size(), stream);

    std::vector<value_t> out_dists_ref_h = params.out_dists_ref_h;

    allocate(out_dists_ref, (indptr_h.size() - 1) * (indptr_h.size() - 1));

    update_device(out_dists_ref, out_dists_ref_h.data(), out_dists_ref_h.size(),
                  stream);
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      SparseDistanceInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    make_data();

    Distance::distances_config_t<value_idx, value_t> dist_config;
    dist_config.b_nrows = params.indptr_h.size() - 1;
    dist_config.b_ncols = params.n_cols;
    dist_config.b_nnz = params.indices_h.size();
    dist_config.b_indptr = indptr;
    dist_config.b_indices = indices;
    dist_config.b_data = data;
    dist_config.a_nrows = params.indptr_h.size() - 1;
    dist_config.a_ncols = params.n_cols;
    dist_config.a_nnz = params.indices_h.size();
    dist_config.a_indptr = indptr;
    dist_config.a_indices = indices;
    dist_config.a_data = data;
    dist_config.handle = cusparseHandle;
    dist_config.allocator = alloc;
    dist_config.stream = stream;

    int out_size = dist_config.a_nrows * dist_config.b_nrows;

    allocate(out_dists, out_size);

    ML::Logger::get().setLevel(CUML_LEVEL_INFO);

    pairwiseDistance(out_dists, dist_config, params.metric);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_dists));
    CUDA_CHECK(hipFree(out_dists_ref));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_dists_ref, out_dists, 16, Compare<value_t>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t cusparseHandle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_t *out_dists, *out_dists_ref;

  SparseDistanceInputs<value_idx, value_t> params;
};

const std::vector<SparseDistanceInputs<int, float>> inputs_i32_f = {
  {2,
   {0, 2, 4, 6, 8},
   {0, 1, 0, 1, 0, 1, 0, 1},  // indices
   {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
   {
     // dense output
     0.0,
     4.0,
     3026.0,
     226.0,
     4.0,
     0.0,
     2930.0,
     234.0,
     3026.0,
     2930.0,
     0.0,
     1832.0,
     226.0,
     234.0,
     1832.0,
     0.0,
   },
   raft::distance::DistanceType::EucExpandedL2},
  {2,
   {0, 2, 4, 6, 8},
   {0, 1, 0, 1, 0, 1, 0, 1},
   {1.0f, 2.0f, 1.0f, 2.0f, 1.0f, 2.0f, 1.0f, 2.0f},
   {5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0,
    5.0},
   raft::distance::DistanceType::InnerProduct

  }};
typedef SparseDistanceTest<int, float> SparseDistanceTestF;
TEST_P(SparseDistanceTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(SparseDistanceTests, SparseDistanceTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Selection
};  // end namespace Sparse
};  // end namespace MLCommon
