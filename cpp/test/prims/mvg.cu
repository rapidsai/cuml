#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <cmath>
#include <iostream>
#include <random/mvg.cuh>
#include <random>
#include "test_utils.h"

// mvg.h takes in matrices that are colomn major (as in fortan)
#define IDX2C(i, j, ld) (j * ld + i)

namespace MLCommon {
namespace Random {

// helper kernels
/// @todo Duplicate called vctwiseAccumulate in utils.h (Kalman Filters,
// i think that is much better to use., more general)
template <typename T>
__global__ void En_KF_accumulate(const int nPoints, const int dim, const T *X,
                                 T *x) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int col = idx % dim;
  int row = idx / dim;
  if (col < dim && row < nPoints) myAtomicAdd(x + col, X[idx]);
}

template <typename T>
__global__ void En_KF_normalize(const int divider, const int dim, T *x) {
  int xi = threadIdx.x + blockDim.x * blockIdx.x;
  if (xi < dim) x[xi] = x[xi] / divider;
}

template <typename T>
__global__ void En_KF_dif(const int nPoints, const int dim, const T *X,
                          const T *x, T *X_diff) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int col = idx % dim;
  int row = idx / dim;
  if (col < dim && row < nPoints) X_diff[idx] = X[idx] - x[col];
}

// for specialising tests
enum Correlation : unsigned char {
  CORRELATED,  // = 0
  UNCORRELATED
};

template <typename T>
struct MVGInputs {
  T tolerance;
  typename MultiVarGaussian<T>::Decomposer method;
  Correlation corr;
  int dim, nPoints;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const MVGInputs<T> &dims) {
  return os;
}

template <typename T>
class MVGTest : public ::testing::TestWithParam<MVGInputs<T>> {
 protected:
  void SetUp() override {
    // getting params
    params = ::testing::TestWithParam<MVGInputs<T>>::GetParam();
    dim = params.dim;
    nPoints = params.nPoints;
    method = params.method;
    corr = params.corr;
    tolerance = params.tolerance;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUDA_CHECK(hipStreamCreate(&stream));

    // preparing to store stuff
    P = (T *)malloc(sizeof(T) * dim * dim);
    x = (T *)malloc(sizeof(T) * dim);
    X = (T *)malloc(sizeof(T) * dim * nPoints);
    CUDA_CHECK(hipMalloc((void **)&P_d, sizeof(T) * dim * dim));
    CUDA_CHECK(hipMalloc((void **)&X_d, sizeof(T) * nPoints * dim));
    CUDA_CHECK(hipMalloc((void **)&x_d, sizeof(T) * dim));
    CUDA_CHECK(hipMalloc((void **)&Rand_cov, sizeof(T) * dim * dim));
    CUDA_CHECK(hipMalloc((void **)&Rand_mean, sizeof(T) * dim));

    // generating random mean and cov.
    srand(params.seed);
    for (int j = 0; j < dim; j++) x[j] = rand() % 100 + 5.0f;

    // for random Cov. martix
    std::default_random_engine generator(params.seed);
    std::uniform_real_distribution<T> distribution(0.0, 1.0);

    // P (developing a +ve definite symm matrix)
    for (int j = 0; j < dim; j++) {
      for (int i = 0; i < j + 1; i++) {
        T k = distribution(generator);
        if (corr == UNCORRELATED) k = 0.0;
        P[IDX2C(i, j, dim)] = k;
        P[IDX2C(j, i, dim)] = k;
        if (i == j) P[IDX2C(i, j, dim)] += dim;
      }
    }

    // porting inputs to gpu
    updateDevice(P_d, P, dim * dim, stream);
    updateDevice(x_d, x, dim, stream);

    // initilizing the mvg
    mvg = new MultiVarGaussian<T>(dim, method);
    size_t o = mvg->init(cublasH, cusolverH, stream);

    // give the workspace area to mvg
    CUDA_CHECK(hipMalloc((void **)&workspace_d, o));
    mvg->set_workspace(workspace_d);

    // get gaussians in X_d | P_d is destroyed.
    mvg->give_gaussian(nPoints, P_d, X_d, x_d);

    // saving the mean of the randoms in Rand_mean
    //@todo can be swapped with a API that calculates mean
    CUDA_CHECK(hipMemset(Rand_mean, 0, dim * sizeof(T)));
    dim3 block = (64);
    dim3 grid = (ceildiv(nPoints * dim, (int)block.x));
    En_KF_accumulate<<<grid, block>>>(nPoints, dim, X_d, Rand_mean);
    CUDA_CHECK(hipPeekAtLastError());
    grid = (ceildiv(dim, (int)block.x));
    En_KF_normalize<<<grid, block>>>(nPoints, dim, Rand_mean);
    CUDA_CHECK(hipPeekAtLastError());

    // storing the error wrt random point mean in X_d
    grid = (ceildiv(dim * nPoints, (int)block.x));
    En_KF_dif<<<grid, block>>>(nPoints, dim, X_d, Rand_mean, X_d);
    CUDA_CHECK(hipPeekAtLastError());

    // finding the cov matrix, placing in Rand_cov
    T alfa = 1.0 / (nPoints - 1), beta = 0.0;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUBLAS_CHECK(raft::linalg::cublasgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, dim,
                                          dim, nPoints, &alfa, X_d, dim, X_d,
                                          dim, &beta, Rand_cov, dim, stream));

    // restoring cov provided into P_d
    updateDevice(P_d, P, dim * dim, stream);
  }

  void TearDown() override {
    // freeing mallocs
    CUDA_CHECK(hipFree(P_d));
    CUDA_CHECK(hipFree(X_d));
    CUDA_CHECK(hipFree(workspace_d));
    free(P);
    free(x);
    free(X);

    // deleting mvg
    mvg->deinit();
    delete mvg;

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  MVGInputs<T> params;
  T *P, *x, *X, *workspace_d, *P_d, *x_d, *X_d;
  int dim, nPoints;
  typename MultiVarGaussian<T>::Decomposer method;
  Correlation corr;
  MultiVarGaussian<T> *mvg = NULL;
  T *Rand_cov, *Rand_mean, tolerance;
  hipblasHandle_t cublasH;
  hipsolverHandle_t cusolverH;
  hipStream_t stream;
};  // end of MVGTest class

///@todo find out the reason that Un-correlated covs are giving problems (in qr)
// Declare your inputs
const std::vector<MVGInputs<float>> inputsf = {
  {0.3f, MultiVarGaussian<float>::Decomposer::chol_decomp,
   Correlation::CORRELATED, 5, 30000, 6ULL},
  {0.1f, MultiVarGaussian<float>::Decomposer::chol_decomp,
   Correlation::UNCORRELATED, 5, 30000, 6ULL},
  {0.25f, MultiVarGaussian<float>::Decomposer::jacobi, Correlation::CORRELATED,
   5, 30000, 6ULL},
  {0.1f, MultiVarGaussian<float>::Decomposer::jacobi, Correlation::UNCORRELATED,
   5, 30000, 6ULL},
  {0.2f, MultiVarGaussian<float>::Decomposer::qr, Correlation::CORRELATED, 5,
   30000, 6ULL},
  // { 0.2f,          MultiVarGaussian<float>::Decomposer::qr,
  // Correlation::UNCORRELATED, 5, 30000, 6ULL}
};
const std::vector<MVGInputs<double>> inputsd = {
  {0.25, MultiVarGaussian<double>::Decomposer::chol_decomp,
   Correlation::CORRELATED, 10, 3000000, 6ULL},
  {0.1, MultiVarGaussian<double>::Decomposer::chol_decomp,
   Correlation::UNCORRELATED, 10, 3000000, 6ULL},
  {0.25, MultiVarGaussian<double>::Decomposer::jacobi, Correlation::CORRELATED,
   10, 3000000, 6ULL},
  {0.1, MultiVarGaussian<double>::Decomposer::jacobi, Correlation::UNCORRELATED,
   10, 3000000, 6ULL},
  {0.2, MultiVarGaussian<double>::Decomposer::qr, Correlation::CORRELATED, 10,
   3000000, 6ULL},
  // { 0.2,          MultiVarGaussian<double>::Decomposer::qr,
  // Correlation::UNCORRELATED, 10, 3000000, 6ULL}
};

// make the tests
typedef MVGTest<float> MVGTestF;
typedef MVGTest<double> MVGTestD;
TEST_P(MVGTestF, MeanIsCorrectF) {
  EXPECT_TRUE(devArrMatch(x_d, Rand_mean, dim, CompareApprox<float>(tolerance)))
    << " in MeanIsCorrect";
}
TEST_P(MVGTestF, CovIsCorrectF) {
  EXPECT_TRUE(
    devArrMatch(P_d, Rand_cov, dim, dim, CompareApprox<float>(tolerance)))
    << " in CovIsCorrect";
}
TEST_P(MVGTestD, MeanIsCorrectD) {
  EXPECT_TRUE(
    devArrMatch(x_d, Rand_mean, dim, CompareApprox<double>(tolerance)))
    << " in MeanIsCorrect";
}
TEST_P(MVGTestD, CovIsCorrectD) {
  EXPECT_TRUE(
    devArrMatch(P_d, Rand_cov, dim, dim, CompareApprox<double>(tolerance)))
    << " in CovIsCorrect";
}

// call the tests
INSTANTIATE_TEST_CASE_P(MVGTests, MVGTestF, ::testing::ValuesIn(inputsf));
INSTANTIATE_TEST_CASE_P(MVGTests, MVGTestD, ::testing::ValuesIn(inputsd));

};  // end of namespace Random
};  // end of namespace MLCommon
