/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <test_utils.h>
#include <iostream>
#include <vector>
#include "cuml/neighbors/knn.hpp"

namespace ML {

using namespace MLCommon;

/**
 *
 * NOTE: Not exhaustively testing the kNN implementation since
 * we are using FAISS for this. Just testing API to verify the
 * knn.cu class is accepting inputs and providing outputs as
 * expected.
 */
template <typename T>
class KNNTest : public ::testing::Test {
 protected:
  void basicTest() {
    // Allocate input
    allocate(d_train_inputs, n * d);

    // Allocate reference arrays
    allocate<long>(d_ref_I, n * n);
    allocate(d_ref_D, n * n);

    // Allocate predicted arrays
    allocate<long>(d_pred_I, n * n);
    allocate(d_pred_D, n * n);

    // make testdata on host
    std::vector<T> h_train_inputs = {1.0, 50.0, 51.0};
    h_train_inputs.resize(n);
    updateDevice(d_train_inputs, h_train_inputs.data(), n * d, 0);

    std::vector<T> h_res_D = {0.0, 49.0, 50.0, 0.0, 1.0, 49.0, 0.0, 1.0, 50.0};
    h_res_D.resize(n * n);
    updateDevice(d_ref_D, h_res_D.data(), n * n, 0);

    std::vector<long> h_res_I = {0, 1, 2, 1, 2, 0, 2, 1, 0};
    h_res_I.resize(n * n);
    updateDevice<long>(d_ref_I, h_res_I.data(), n * n, 0);

    std::vector<float *> ptrs(1);
    std::vector<int> sizes(1);
    ptrs[0] = d_train_inputs;
    sizes[0] = n;

    knn->fit(ptrs, sizes);
    knn->search(d_train_inputs, n, d_pred_I, d_pred_D, n);
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    CUDA_CHECK(hipFree(d_train_inputs));
    CUDA_CHECK(hipFree(d_pred_I));
    CUDA_CHECK(hipFree(d_pred_D));
    CUDA_CHECK(hipFree(d_ref_I));
    CUDA_CHECK(hipFree(d_ref_D));
  }

 protected:
  T *d_train_inputs;

  int n = 3;
  int d = 1;

  long *d_pred_I;
  T *d_pred_D;

  long *d_ref_I;
  T *d_ref_D;

  cumlHandle handle;
  kNN *knn = new kNN(handle, d);
};

typedef KNNTest<float> KNNTestF;
TEST_F(KNNTestF, Fit) {
  ASSERT_TRUE(devArrMatch(d_ref_D, d_pred_D, n * n, Compare<float>()));
  ASSERT_TRUE(devArrMatch(d_ref_I, d_pred_I, n * n, Compare<long>()));
}

}  // end namespace ML
