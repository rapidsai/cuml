/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <glm/ols.cuh>
#include <glm/ridge.cuh>
#include <raft/cuda_utils.cuh>

namespace ML {
namespace GLM {

using namespace MLCommon;

template <typename T>
struct RidgeInputs {
  T tol;
  int n_row;
  int n_col;
  int n_row_2;
  int algo;
  T alpha;
};

template <typename T>
class RidgeTest : public ::testing::TestWithParam<RidgeInputs<T>> {
 protected:
  void basicTest() {
    params = ::testing::TestWithParam<RidgeInputs<T>>::GetParam();
    int len = params.n_row * params.n_col;
    int len2 = params.n_row_2 * params.n_col;

    raft::allocate(data, len);
    raft::allocate(labels, params.n_row);
    raft::allocate(coef, params.n_col);
    raft::allocate(coef2, params.n_col);
    raft::allocate(coef3, params.n_col);
    raft::allocate(coef_ref, params.n_col);
    raft::allocate(coef2_ref, params.n_col);
    raft::allocate(coef3_ref, params.n_col);
    raft::allocate(pred_data, len2);
    raft::allocate(pred, params.n_row_2);
    raft::allocate(pred_ref, params.n_row_2);
    raft::allocate(pred2, params.n_row_2);
    raft::allocate(pred2_ref, params.n_row_2);
    raft::allocate(pred3, params.n_row_2);
    raft::allocate(pred3_ref, params.n_row_2);
    T alpha = params.alpha;

    T data_h[len] = {0.0, 0.0, 1.0, 0.0, 0.0, 1.0};
    raft::update_device(data, data_h, len, stream);

    T labels_h[params.n_row] = {0.0, 0.1, 1.0};
    raft::update_device(labels, labels_h, params.n_row, stream);

    T coef_ref_h[params.n_col] = {0.39999998, 0.4};
    raft::update_device(coef_ref, coef_ref_h, params.n_col, stream);

    T coef2_ref_h[params.n_col] = {0.3454546, 0.34545454};
    raft::update_device(coef2_ref, coef2_ref_h, params.n_col, stream);

    T coef3_ref_h[params.n_col] = {0.3799999, 0.38000008};
    raft::update_device(coef3_ref, coef3_ref_h, params.n_col, stream);

    T pred_data_h[len2] = {0.5, 2.0, 0.2, 1.0};
    raft::update_device(pred_data, pred_data_h, len2, stream);

    T pred_ref_h[params.n_row_2] = {0.28, 1.1999999};
    raft::update_device(pred_ref, pred_ref_h, params.n_row_2, stream);

    T pred2_ref_h[params.n_row_2] = {0.37818184, 1.1727273};
    raft::update_device(pred2_ref, pred2_ref_h, params.n_row_2, stream);

    T pred3_ref_h[params.n_row_2] = {0.37933332, 1.2533332};
    raft::update_device(pred3_ref, pred3_ref_h, params.n_row_2, stream);

    intercept = T(0);

    ridgeFit(handle, data, params.n_row, params.n_col, labels, &alpha, 1, coef,
             &intercept, false, false, stream, params.algo);

    gemmPredict(handle, pred_data, params.n_row_2, params.n_col, coef,
                intercept, pred, stream);

    raft::update_device(data, data_h, len, stream);
    raft::update_device(labels, labels_h, params.n_row, stream);

    intercept2 = T(0);
    ridgeFit(handle, data, params.n_row, params.n_col, labels, &alpha, 1, coef2,
             &intercept2, true, false, stream, params.algo);

    gemmPredict(handle, pred_data, params.n_row_2, params.n_col, coef2,
                intercept2, pred2, stream);

    raft::update_device(data, data_h, len, stream);
    raft::update_device(labels, labels_h, params.n_row, stream);

    intercept3 = T(0);
    ridgeFit(handle, data, params.n_row, params.n_col, labels, &alpha, 1, coef3,
             &intercept3, true, true, stream, params.algo);

    gemmPredict(handle, pred_data, params.n_row_2, params.n_col, coef3,
                intercept3, pred3, stream);
  }

  void basicTest2() {
    params = ::testing::TestWithParam<RidgeInputs<T>>::GetParam();
    int len = params.n_row * params.n_col;

    raft::allocate(data_sc, len);
    raft::allocate(labels_sc, len);
    raft::allocate(coef_sc, 1);
    raft::allocate(coef_sc_ref, 1);

    std::vector<T> data_h = {1.0, 1.0, 2.0, 2.0, 1.0, 2.0};
    data_h.resize(len);
    raft::update_device(data_sc, data_h.data(), len, stream);

    std::vector<T> labels_h = {6.0, 8.0, 9.0, 11.0, -1.0, 2.0};
    labels_h.resize(len);
    raft::update_device(labels_sc, labels_h.data(), len, stream);

    std::vector<T> coef_sc_ref_h = {1.8};
    coef_sc_ref_h.resize(1);
    raft::update_device(coef_sc_ref, coef_sc_ref_h.data(), 1, stream);

    T intercept_sc = T(0);
    T alpha_sc = T(1.0);

    ridgeFit(handle, data_sc, len, 1, labels_sc, &alpha_sc, 1, coef_sc,
             &intercept_sc, true, false, stream, params.algo);
  }

  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);
    basicTest();
    basicTest2();
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(coef));
    CUDA_CHECK(hipFree(coef_ref));
    CUDA_CHECK(hipFree(coef2));
    CUDA_CHECK(hipFree(coef2_ref));
    CUDA_CHECK(hipFree(coef3));
    CUDA_CHECK(hipFree(coef3_ref));
    CUDA_CHECK(hipFree(pred_data));
    CUDA_CHECK(hipFree(pred));
    CUDA_CHECK(hipFree(pred_ref));
    CUDA_CHECK(hipFree(pred2));
    CUDA_CHECK(hipFree(pred2_ref));
    CUDA_CHECK(hipFree(pred3));
    CUDA_CHECK(hipFree(pred3_ref));

    CUDA_CHECK(hipFree(data_sc));
    CUDA_CHECK(hipFree(labels_sc));
    CUDA_CHECK(hipFree(coef_sc));
    CUDA_CHECK(hipFree(coef_sc_ref));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  RidgeInputs<T> params;
  T *data, *labels, *coef, *coef_ref, *pred_data, *pred, *pred_ref;
  T *coef2, *coef2_ref, *pred2, *pred2_ref;
  T *coef3, *coef3_ref, *pred3, *pred3_ref;
  T *data_sc, *labels_sc, *coef_sc, *coef_sc_ref;
  T intercept, intercept2, intercept3;
  raft::handle_t handle;
  hipStream_t stream;
};

const std::vector<RidgeInputs<float>> inputsf2 = {{0.001f, 3, 2, 2, 0, 0.5f},
                                                  {0.001f, 3, 2, 2, 1, 0.5f}};

const std::vector<RidgeInputs<double>> inputsd2 = {{0.001, 3, 2, 2, 0, 0.5},
                                                   {0.001, 3, 2, 2, 1, 0.5}};

typedef RidgeTest<float> RidgeTestF;
TEST_P(RidgeTestF, Fit) {
  ASSERT_TRUE(raft::devArrMatch(coef_ref, coef, params.n_col,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef2_ref, coef2, params.n_col,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef3_ref, coef3, params.n_col,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred_ref, pred, params.n_row_2,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred2_ref, pred2, params.n_row_2,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred3_ref, pred3, params.n_row_2,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef_sc_ref, coef_sc, 1,
                                raft::CompareApproxAbs<float>(params.tol)));
}

typedef RidgeTest<double> RidgeTestD;
TEST_P(RidgeTestD, Fit) {
  ASSERT_TRUE(raft::devArrMatch(coef_ref, coef, params.n_col,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef2_ref, coef2, params.n_col,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef3_ref, coef3, params.n_col,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred_ref, pred, params.n_row_2,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred2_ref, pred2, params.n_row_2,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred3_ref, pred3, params.n_row_2,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef_sc_ref, coef_sc, 1,
                                raft::CompareApproxAbs<double>(params.tol)));
}

INSTANTIATE_TEST_CASE_P(RidgeTests, RidgeTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(RidgeTests, RidgeTestD, ::testing::ValuesIn(inputsd2));

}  // namespace GLM
}  // end namespace ML
