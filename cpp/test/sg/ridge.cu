/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <test_utils.h>
#include <cuda_utils.cuh>
#include <glm/ridge.cuh>

namespace ML {
namespace GLM {

using namespace MLCommon;

template <typename T>
struct RidgeInputs {
  T tol;
  int n_row;
  int n_col;
  int n_row_2;
  int algo;
  T alpha;
};

template <typename T>
class RidgeTest : public ::testing::TestWithParam<RidgeInputs<T>> {
 protected:
  void basicTest() {
    params = ::testing::TestWithParam<RidgeInputs<T>>::GetParam();
    int len = params.n_row * params.n_col;
    int len2 = params.n_row_2 * params.n_col;

    allocate(data, len);
    allocate(labels, params.n_row);
    allocate(coef, params.n_col);
    allocate(coef2, params.n_col);
    allocate(coef3, params.n_col);
    allocate(coef_ref, params.n_col);
    allocate(coef2_ref, params.n_col);
    allocate(coef3_ref, params.n_col);
    allocate(pred_data, len2);
    allocate(pred, params.n_row_2);
    allocate(pred_ref, params.n_row_2);
    allocate(pred2, params.n_row_2);
    allocate(pred2_ref, params.n_row_2);
    allocate(pred3, params.n_row_2);
    allocate(pred3_ref, params.n_row_2);
    T alpha = params.alpha;

    T data_h[len] = {0.0, 0.0, 1.0, 0.0, 0.0, 1.0};
    updateDevice(data, data_h, len, stream);

    T labels_h[params.n_row] = {0.0, 0.1, 1.0};
    updateDevice(labels, labels_h, params.n_row, stream);

    T coef_ref_h[params.n_col] = {0.39999998, 0.4};
    updateDevice(coef_ref, coef_ref_h, params.n_col, stream);

    T coef2_ref_h[params.n_col] = {0.3454546, 0.34545454};
    updateDevice(coef2_ref, coef2_ref_h, params.n_col, stream);

    T coef3_ref_h[params.n_col] = {0.3799999, 0.38000008};
    updateDevice(coef3_ref, coef3_ref_h, params.n_col, stream);

    T pred_data_h[len2] = {0.5, 2.0, 0.2, 1.0};
    updateDevice(pred_data, pred_data_h, len2, stream);

    T pred_ref_h[params.n_row_2] = {0.28, 1.1999999};
    updateDevice(pred_ref, pred_ref_h, params.n_row_2, stream);

    T pred2_ref_h[params.n_row_2] = {0.37818184, 1.1727273};
    updateDevice(pred2_ref, pred2_ref_h, params.n_row_2, stream);

    T pred3_ref_h[params.n_row_2] = {0.37933332, 1.2533332};
    updateDevice(pred3_ref, pred3_ref_h, params.n_row_2, stream);

    intercept = T(0);

    ridgeFit(handle, data, params.n_row, params.n_col, labels, &alpha, 1, coef,
             &intercept, false, false, stream, params.algo);

    ridgePredict(handle, pred_data, params.n_row_2, params.n_col, coef,
                 intercept, pred, stream);

    updateDevice(data, data_h, len, stream);
    updateDevice(labels, labels_h, params.n_row, stream);

    intercept2 = T(0);
    ridgeFit(handle, data, params.n_row, params.n_col, labels, &alpha, 1, coef2,
             &intercept2, true, false, stream, params.algo);

    ridgePredict(handle, pred_data, params.n_row_2, params.n_col, coef2,
                 intercept2, pred2, stream);

    updateDevice(data, data_h, len, stream);
    updateDevice(labels, labels_h, params.n_row, stream);

    intercept3 = T(0);
    ridgeFit(handle, data, params.n_row, params.n_col, labels, &alpha, 1, coef3,
             &intercept3, true, true, stream, params.algo);

    ridgePredict(handle, pred_data, params.n_row_2, params.n_col, coef3,
                 intercept3, pred3, stream);
  }

  void basicTest2() {
    params = ::testing::TestWithParam<RidgeInputs<T>>::GetParam();
    int len = params.n_row * params.n_col;

    allocate(data_sc, len);
    allocate(labels_sc, len);
    allocate(coef_sc, 1);
    allocate(coef_sc_ref, 1);

    std::vector<T> data_h = {1.0, 1.0, 2.0, 2.0, 1.0, 2.0};
    data_h.resize(len);
    updateDevice(data_sc, data_h.data(), len, stream);

    std::vector<T> labels_h = {6.0, 8.0, 9.0, 11.0, -1.0, 2.0};
    labels_h.resize(len);
    updateDevice(labels_sc, labels_h.data(), len, stream);

    std::vector<T> coef_sc_ref_h = {1.8};
    coef_sc_ref_h.resize(1);
    updateDevice(coef_sc_ref, coef_sc_ref_h.data(), 1, stream);

    T intercept_sc = T(0);
    T alpha_sc = T(1.0);

    ridgeFit(handle, data_sc, len, 1, labels_sc, &alpha_sc, 1, coef_sc,
             &intercept_sc, true, false, stream, params.algo);
  }

  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);
    basicTest();
    basicTest2();
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(coef));
    CUDA_CHECK(hipFree(coef_ref));
    CUDA_CHECK(hipFree(coef2));
    CUDA_CHECK(hipFree(coef2_ref));
    CUDA_CHECK(hipFree(coef3));
    CUDA_CHECK(hipFree(coef3_ref));
    CUDA_CHECK(hipFree(pred_data));
    CUDA_CHECK(hipFree(pred));
    CUDA_CHECK(hipFree(pred_ref));
    CUDA_CHECK(hipFree(pred2));
    CUDA_CHECK(hipFree(pred2_ref));
    CUDA_CHECK(hipFree(pred3));
    CUDA_CHECK(hipFree(pred3_ref));

    CUDA_CHECK(hipFree(data_sc));
    CUDA_CHECK(hipFree(labels_sc));
    CUDA_CHECK(hipFree(coef_sc));
    CUDA_CHECK(hipFree(coef_sc_ref));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  RidgeInputs<T> params;
  T *data, *labels, *coef, *coef_ref, *pred_data, *pred, *pred_ref;
  T *coef2, *coef2_ref, *pred2, *pred2_ref;
  T *coef3, *coef3_ref, *pred3, *pred3_ref;
  T *data_sc, *labels_sc, *coef_sc, *coef_sc_ref;
  T intercept, intercept2, intercept3;
  raft::handle_t handle;
  hipStream_t stream;
};

const std::vector<RidgeInputs<float>> inputsf2 = {{0.001f, 3, 2, 2, 0, 0.5f},
                                                  {0.001f, 3, 2, 2, 1, 0.5f}};

const std::vector<RidgeInputs<double>> inputsd2 = {{0.001, 3, 2, 2, 0, 0.5},
                                                   {0.001, 3, 2, 2, 1, 0.5}};

typedef RidgeTest<float> RidgeTestF;
TEST_P(RidgeTestF, Fit) {
  ASSERT_TRUE(devArrMatch(coef_ref, coef, params.n_col,
                          CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(devArrMatch(coef2_ref, coef2, params.n_col,
                          CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(devArrMatch(coef3_ref, coef3, params.n_col,
                          CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(devArrMatch(pred_ref, pred, params.n_row_2,
                          CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(devArrMatch(pred2_ref, pred2, params.n_row_2,
                          CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(devArrMatch(pred3_ref, pred3, params.n_row_2,
                          CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(
    devArrMatch(coef_sc_ref, coef_sc, 1, CompareApproxAbs<float>(params.tol)));
}

typedef RidgeTest<double> RidgeTestD;
TEST_P(RidgeTestD, Fit) {
  ASSERT_TRUE(devArrMatch(coef_ref, coef, params.n_col,
                          CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(devArrMatch(coef2_ref, coef2, params.n_col,
                          CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(devArrMatch(coef3_ref, coef3, params.n_col,
                          CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(devArrMatch(pred_ref, pred, params.n_row_2,
                          CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(devArrMatch(pred2_ref, pred2, params.n_row_2,
                          CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(devArrMatch(pred3_ref, pred3, params.n_row_2,
                          CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(
    devArrMatch(coef_sc_ref, coef_sc, 1, CompareApproxAbs<double>(params.tol)));
}

INSTANTIATE_TEST_CASE_P(RidgeTests, RidgeTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(RidgeTests, RidgeTestD, ::testing::ValuesIn(inputsd2));

}  // namespace GLM
}  // end namespace ML
