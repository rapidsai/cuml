/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <linalg/transpose.h>
#include <test_utils.h>
#include <cuda_utils.cuh>
#include <cuml/datasets/make_blobs.hpp>
#include <cuml/ensemble/randomforest.hpp>

namespace ML {

using namespace MLCommon;

struct RfInputs {
  int n_rows;
  int n_cols;
  int n_trees;
  float max_features;
  float rows_sample;
  int max_depth;
  int max_leaves;
  bool bootstrap;
  bool bootstrap_features;
  int n_bins;
  int split_algo;
  int min_rows_per_node;
  float min_impurity_decrease;
  int n_streams;
  CRITERION split_criterion;
};

template <typename T>
class RFBatchedClsTest : public ::testing::TestWithParam<RfInputs> {
 protected:
  void basicTest() {
    params = ::testing::TestWithParam<RfInputs>::GetParam();

    DecisionTree::DecisionTreeParams tree_params;
    set_tree_params(tree_params, params.max_depth, params.max_leaves,
                    params.max_features, params.n_bins, params.split_algo,
                    params.min_rows_per_node, params.min_impurity_decrease,
                    params.bootstrap_features, params.split_criterion, false,
                    true);
    RF_params rf_params;
    set_all_rf_params(rf_params, params.n_trees, params.bootstrap,
                      params.rows_sample, -1, params.n_streams, tree_params);

    CUDA_CHECK(hipStreamCreate(&stream));
    handle.reset(new raft::handle_t(rf_params.n_streams));
    handle->set_stream(stream);
    auto allocator = handle->get_device_allocator();

    int data_len = params.n_rows * params.n_cols;
    data = (T*)allocator->allocate(data_len * sizeof(T), stream);
    labels = (int*)allocator->allocate(params.n_rows * sizeof(int), stream);
    predicted_labels =
      (int*)allocator->allocate(params.n_rows * sizeof(int), stream);

    Datasets::make_blobs(*handle, data, labels, params.n_rows, params.n_cols, 5,
                         false, nullptr, nullptr, T(0.1), false, T(-0.5),
                         T(0.5), 3536699ULL);

    labels_h.resize(params.n_rows);
    raft::update_host(labels_h.data(), labels, params.n_rows, stream);
    preprocess_labels(params.n_rows, labels_h, labels_map);
    raft::update_device(labels, labels_h.data(), params.n_rows, stream);

    // Training part
    forest = new typename ML::RandomForestMetaData<T, int>;
    null_trees_ptr(forest);
    fit(*handle, forest, data, params.n_rows, params.n_cols, labels,
        labels_map.size(), rf_params);

    // predict function expects row major lay out of data, so we need to
    // transpose the data first
    T* data_row_major;
    data_row_major = (T*)allocator->allocate(data_len * sizeof(T), stream);
    hipblasHandle_t hipblas.h = handle->get_cublas_handle();
    raft::linalg::transpose(*handle, data, data_row_major, params.n_rows,
                            params.n_cols, stream);

    predict(*handle, forest, data_row_major, params.n_rows, params.n_cols,
            predicted_labels);
    raft::update_host(labels_h.data(), predicted_labels, params.n_rows, stream);

    RF_metrics tmp =
      score(*handle, forest, labels, params.n_rows, predicted_labels);

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    accuracy = tmp.accuracy;
    allocator->deallocate(data_row_major, data_len * sizeof(T), stream);
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    auto allocator = handle->get_device_allocator();
    accuracy = -1.0f;
    postprocess_labels(params.n_rows, labels_h, labels_map);
    labels_h.clear();
    labels_map.clear();

    allocator->deallocate(labels, params.n_rows * sizeof(int), stream);
    allocator->deallocate(predicted_labels, params.n_rows * sizeof(int),
                          stream);
    allocator->deallocate(data, params.n_rows * params.n_cols * sizeof(T),
                          stream);
    delete forest;
    handle.reset();
  }

 protected:
  std::shared_ptr<raft::handle_t> handle;
  hipStream_t stream;
  RfInputs params;
  T* data;
  int* labels;
  std::vector<int> labels_h;
  std::map<int, int>
    labels_map;  //unique map of labels to int vals starting from 0

  RandomForestMetaData<T, int>* forest;
  float accuracy = -1.0f;  // overriden in each test SetUp and TearDown

  int* predicted_labels;
};

//-------------------------------------------------------------------------------------------------------------------------------------
const std::vector<RfInputs> inputsf2_clf = {
  {20000, 10, 25, 1.0f, 0.4f, 16, -1, true, false, 10,
   SPLIT_ALGO::GLOBAL_QUANTILE, 2, 0.0, 2, CRITERION::GINI},
  {20000, 10, 5, 1.0f, 0.4f, 14, -1, true, false, 10,
   SPLIT_ALGO::GLOBAL_QUANTILE, 2, 0.0, 2, CRITERION::ENTROPY}};

typedef RFBatchedClsTest<float> RFBatchedClsTestF;
TEST_P(RFBatchedClsTestF, Fit) {
  if (!params.bootstrap && (params.max_features == 1.0f)) {
    ASSERT_TRUE(accuracy == 1.0f);
  } else {
    ASSERT_TRUE(accuracy >= 0.75f);  // Empirically derived accuracy range
  }
}

INSTANTIATE_TEST_CASE_P(RFBatchedClsTests, RFBatchedClsTestF,
                        ::testing::ValuesIn(inputsf2_clf));

typedef RFBatchedClsTest<double> RFBatchedClsTestD;
TEST_P(RFBatchedClsTestD, Fit) {
  if (!params.bootstrap && (params.max_features == 1.0f)) {
    ASSERT_TRUE(accuracy == 1.0f);
  } else {
    ASSERT_TRUE(accuracy >= 0.75f);  // Empirically derived accuracy range
  }
}

INSTANTIATE_TEST_CASE_P(RFBatchedClsTests, RFBatchedClsTestD,
                        ::testing::ValuesIn(inputsf2_clf));

}  // end namespace ML
