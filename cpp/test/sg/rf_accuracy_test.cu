/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <cuml/ensemble/randomforest.hpp>
#include <raft/cuda_utils.cuh>
#include <raft/random/rng.cuh>

namespace ML {

using namespace MLCommon;

struct RFInputs {
  int n_rows_train;
  int n_rows_test;
  uint64_t seed;
  int n_reps;
  float pct_zero_class;
  float min_expected_acc;
};

template <typename T>
class RFClassifierAccuracyTest : public ::testing::TestWithParam<RFInputs> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<RFInputs>::GetParam();
    rng.reset(new raft::random::Rng(params.seed));
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.reset(new raft::handle_t(1));
    handle->set_stream(stream);
    auto allocator = handle->get_device_allocator();
    setRFParams();
    X_train = (T *)allocator->allocate(params.n_rows_train * sizeof(T), stream);
    y_train =
      (int *)allocator->allocate(params.n_rows_train * sizeof(int), stream);
    X_test = (T *)allocator->allocate(params.n_rows_test * sizeof(T), stream);
    y_test =
      (int *)allocator->allocate(params.n_rows_test * sizeof(int), stream);
    y_pred =
      (int *)allocator->allocate(params.n_rows_test * sizeof(int), stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto allocator = handle->get_device_allocator();
    allocator->deallocate(X_train, params.n_rows_train * sizeof(T), stream);
    allocator->deallocate(y_train, params.n_rows_train * sizeof(int), stream);
    allocator->deallocate(X_test, params.n_rows_test * sizeof(T), stream);
    allocator->deallocate(y_test, params.n_rows_test * sizeof(int), stream);
    allocator->deallocate(y_pred, params.n_rows_test * sizeof(int), stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    handle.reset();
    rng.reset();
  }

  void runTest() {
    for (int i = 0; i < params.n_reps; ++i) {
      loadData(X_train, y_train, params.n_rows_train, 1);
      loadData(X_test, y_test, params.n_rows_test, 1);
      CUDA_CHECK(hipStreamSynchronize(stream));
      auto accuracy = runTrainAndTest();
      ASSERT_GT(accuracy, params.min_expected_acc) << " @repetition=" << i;
    }
  }

 private:
  void setRFParams() {
    auto algo = SPLIT_ALGO::GLOBAL_QUANTILE;
    auto sc = CRITERION::CRITERION_END;

    rfp = set_rf_params(0,     /*max_depth */
                        -1,    /* max_leaves */
                        1.0,   /* max_features */
                        16,    /* n_bins */
                        algo,  /* split_algo */
                        2,     /* min_samples_leaf */
                        2,     /* min_samples_split */
                        0.f,   /* min_impurity_decrease */
                        false, /* bootstrap_features */
                        true,  /* bootstrap */
                        1,     /* n_trees */
                        1.0,   /* max_samples */
                        0,     /* seed */
                        sc,    /* split_criterion */
                        false, /* quantile_per_tree */
                        1,     /* n_streams */
                        false, /* use_experimental_backend */
                        128    /* max_batch_size */
    );
  }

  void loadData(T *X, int *y, int nrows, int ncols) {
    rng->uniform(X, nrows * ncols, T(-1.0), T(1.0), stream);
    rng->bernoulli<float, int>(y, nrows, params.pct_zero_class, stream);
  }

  float runTrainAndTest() {
    auto *forest = new RandomForestMetaData<T, int>;
    forest->trees = nullptr;
    auto &h = *(handle.get());
    fit(h, forest, X_train, params.n_rows_train, 1, y_train, 2, rfp);
    CUDA_CHECK(hipStreamSynchronize(stream));
    predict(h, forest, X_test, params.n_rows_test, 1, y_pred);
    auto metrics = score(h, forest, y_test, params.n_rows_test, y_pred);
    delete forest;
    return metrics.accuracy;
  }

  RFInputs params;
  RF_params rfp;
  std::shared_ptr<raft::handle_t> handle;
  hipStream_t stream;
  T *X_train, *X_test;
  int *y_train, *y_test, *y_pred;
  std::shared_ptr<raft::random::Rng> rng;
};

const std::vector<RFInputs> inputs = {
  {800, 200, 12345ULL, 40, 0.5f, 0.4f},  {800, 200, 12345ULL, 40, 0.8f, 0.7f},
  {800, 200, 67890ULL, 40, 0.5f, 0.4f},  {800, 200, 67890ULL, 40, 0.8f, 0.7f},
  {1000, 250, 67890ULL, 40, 0.9f, 0.8f}, {1000, 250, 67890ULL, 40, 0.1f, 0.8f},
};

#define DEFINE_TEST(clz, name, testName, params) \
  typedef clz name;                              \
  TEST_P(name, Test) { runTest(); }              \
  INSTANTIATE_TEST_CASE_P(testName, name, ::testing::ValuesIn(params))

DEFINE_TEST(RFClassifierAccuracyTest<float>, ClsTestF, RFAccuracy, inputs);
DEFINE_TEST(RFClassifierAccuracyTest<double>, ClsTestD, RFAccuracy, inputs);

#undef DEFINE_TEST

}  // end namespace ML
