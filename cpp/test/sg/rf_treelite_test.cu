/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <decisiontree/decisiontree_impl.h>
#include <decisiontree/treelite_util.h>
#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/gemv.h>
#include <raft/linalg/transpose.h>
#include <sys/stat.h>
#include <test_utils.h>
#include <treelite/c_api.h>
#include <treelite/c_api_runtime.h>
#include <cstdlib>
#include <cuml/ensemble/randomforest.hpp>
#include <fstream>
#include <iostream>
#include <limits>
#include <raft/cuda_utils.cuh>
#include <raft/random/rng.cuh>
#include <string>

namespace ML {

using namespace MLCommon;

template <typename T>  // template useless for now.
struct RfInputs {
  int n_rows;
  int n_cols;
  int n_trees;
  float max_features;
  float max_samples;
  int n_inference_rows;
  int max_depth;
  int max_leaves;
  bool bootstrap;
  bool bootstrap_features;
  int n_bins;
  int split_algo;
  int min_samples_leaf;
  int min_samples_split;
  float min_impurity_decrease;
  int n_streams;
  CRITERION split_criterion;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const RfInputs<T> &dims) {
  return os;
}

template <typename T, typename L>
class RfTreeliteTestCommon : public ::testing::TestWithParam<RfInputs<T>> {
 protected:
  void ConcatenateTreeliteModels() {
    // Test the implementation for converting fitted forest into treelite format.
    ModelHandle concatenated_forest_handle;
    concatenated_forest_handle = concatenate_trees(treelite_indiv_handles);
    compare_concat_forest_to_subforests(concatenated_forest_handle,
                                        treelite_indiv_handles);
    std::string test_name =
      ::testing::UnitTest::GetInstance()->current_test_info()->name();
    // Get the test index from Google current_test_info.
    // The test index is the string after '/' in test_name.
    std::string index_str =
      test_name.substr(test_name.find("/") + 1, test_name.length());

    // Create a directory if the test is the first one in the test case.
    int mkdir_ret = mkdir(test_dir.c_str(), 0700);
    if (mkdir_ret != 0) {
      // Ignore the error if the error is caused by EEXIST.
      // Treelite will generate errors when the directory is not accessible.
      ASSERT(errno == EEXIST, "Call mkdir %s fails.", test_dir.c_str());
    }

    // Create a sub-directory for the test case.
    dir_name = test_dir + index_str;

    CompilerHandle compiler;
    // "ast_navive" is the default compiler treelite used in their Python code.
    TREELITE_CHECK(TreeliteCompilerCreate("ast_native", &compiler));

    int verbose = 0;
    // Generate C code in the directory specified below.
    // The parallel comilplation is disabled. To enable it, one needs to specify parallel_comp of CompilerHandle.
    // Treelite will create a directory if it doesn't exist.
    TREELITE_CHECK(TreeliteCompilerGenerateCode(
      compiler, treelite_indiv_handles[0], verbose, dir_name.c_str()));
    TREELITE_CHECK(TreeliteCompilerFree(compiler));

    // Options copied from
    // https://github.com/dmlc/treelite/blob/528d883f8f39eb5dd633e929b95915b63e210b39/python/treelite/contrib/__init__.py.
    std::string obj_cmd = "gcc -c -O3 -o " + dir_name + "/main.o " + dir_name +
                          "/main.c -fPIC "
                          "-std=c99 -lm";

    std::string lib_cmd = "gcc -shared -O3 -o " + dir_name +
                          "/treelite_model.so " + dir_name +
                          "/main.o -std=c99 -lm";

    ASSERT(system(obj_cmd.c_str()) == 0, "Call %s fails.", obj_cmd.c_str());
    ASSERT(system(lib_cmd.c_str()) == 0, "Call %s fails.", lib_cmd.c_str());

    PredictorHandle predictor;
    std::string lib_path = dir_name + "/treelite_model.so";

    // -1 means use maximum possible worker threads.
    int worker_thread = -1;
    TREELITE_CHECK(
      TreelitePredictorLoad(lib_path.c_str(), worker_thread, &predictor));

    DMatrixHandle dmat;
    // Current RF doesn't seem to support missing value, put NaN to be safe.
    T missing_value = std::numeric_limits<T>::quiet_NaN();
    TREELITE_CHECK(TreeliteDMatrixCreateFromMat(
      inference_data_h.data(), ML::DecisionTree::TreeliteType<T>::value,
      params.n_inference_rows, params.n_cols, &missing_value, &dmat));

    // Use dense batch so batch_sparse is 0.
    // pred_margin = true means to produce raw margins rather than transformed probability.
    bool pred_margin = false;
    // Allocate larger array for treelite predicted label with using multi-class classification to avoid seg faults.
    // Altough later we only use first params.n_inference_rows elements.
    size_t treelite_predicted_labels_size;

    TREELITE_CHECK(TreelitePredictorPredictBatch(
      predictor, dmat, verbose, pred_margin, treelite_predicted_labels.data(),
      &treelite_predicted_labels_size));

    TREELITE_CHECK(TreeliteDMatrixFree(dmat));
    TREELITE_CHECK(TreelitePredictorFree(predictor));
    TREELITE_CHECK(TreeliteFreeModel(concatenated_forest_handle));
    TREELITE_CHECK(TreeliteFreeModel(treelite_indiv_handles[0]));
    TREELITE_CHECK(TreeliteFreeModel(treelite_indiv_handles[1]));
    TREELITE_CHECK(TreeliteFreeModel(treelite_indiv_handles[2]));
  }

  void getResultAndCheck() {
    // Predict and compare against known labels
    predict(*handle, forest, inference_data_d, params.n_inference_rows,
            params.n_cols, predicted_labels_d);
    RF_metrics tmp = score(*handle, forest, labels_d, params.n_inference_rows,
                           predicted_labels_d);

    CUDA_CHECK(hipStreamSynchronize(stream));

    predicted_labels_h.resize(params.n_inference_rows);
    ref_predicted_labels.resize(params.n_inference_rows);

    raft::update_host(predicted_labels_h.data(), predicted_labels_d,
                      params.n_inference_rows, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i = 0; i < params.n_inference_rows; i++) {
      if (is_classification) {
        ref_predicted_labels[i] = static_cast<float>(predicted_labels_h[i]);
        treelite_predicted_labels[i] =
          treelite_predicted_labels[i] >= 0.5 ? 1 : 0;
      } else {
        ref_predicted_labels[i] = static_cast<float>(predicted_labels_h[i]);
      }
    }

    EXPECT_TRUE(raft::devArrMatchHost(
      ref_predicted_labels.data(), treelite_predicted_labels.data(),
      params.n_inference_rows, raft::Compare<float>(), stream));
  }

  void SetUp() override {
    params = ::testing::TestWithParam<RfInputs<T>>::GetParam();

    rf_params = set_rf_params(
      params.max_depth, params.max_leaves, params.max_features, params.n_bins,
      params.split_algo, params.min_samples_leaf, params.min_samples_split,
      params.min_impurity_decrease, params.bootstrap_features, params.bootstrap,
      params.n_trees, params.max_samples, 0, params.split_criterion, false,
      params.n_streams, false, 128);

    handle.reset(new raft::handle_t(rf_params.n_streams));

    data_len = params.n_rows * params.n_cols;
    inference_data_len = params.n_inference_rows * params.n_cols;

    raft::allocate(data_d, data_len);
    raft::allocate(inference_data_d, inference_data_len);

    raft::allocate(labels_d, params.n_rows);
    raft::allocate(predicted_labels_d, params.n_inference_rows);

    treelite_predicted_labels.resize(params.n_inference_rows);
    ref_predicted_labels.resize(params.n_inference_rows);

    CUDA_CHECK(hipStreamCreate(&stream));
    handle->set_stream(stream);

    forest = new typename ML::RandomForestMetaData<T, L>;
    null_trees_ptr(forest);
    forest_2 = new typename ML::RandomForestMetaData<T, L>;
    null_trees_ptr(forest_2);
    forest_3 = new typename ML::RandomForestMetaData<T, L>;
    null_trees_ptr(forest_3);
    all_forest_info = {forest, forest_2, forest_3};
    data_h.resize(data_len);
    inference_data_h.resize(inference_data_len);

    // Random number generator.
    raft::random::Rng r1(1234ULL);
    // Generate data_d is in column major order.
    r1.uniform(data_d, data_len, T(0.0), T(10.0), stream);
    raft::random::Rng r2(4321ULL);
    // Generate inference_data_d which is in row major order.
    r2.uniform(inference_data_d, inference_data_len, T(0.0), T(10.0), stream);

    raft::update_host(data_h.data(), data_d, data_len, stream);
    raft::update_host(inference_data_h.data(), inference_data_d,
                      inference_data_len, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipFree(data_d));
    CUDA_CHECK(hipFree(inference_data_d));
    CUDA_CHECK(hipFree(labels_d));
    CUDA_CHECK(hipFree(predicted_labels_d));

    delete forest;
    delete forest_2;
    delete forest_3;
    all_forest_info.clear();
    labels_h.clear();
    predicted_labels_h.clear();
    data_h.clear();
    inference_data_h.clear();
    treelite_predicted_labels.clear();
    ref_predicted_labels.clear();
    treelite_indiv_handles.clear();
  }

 protected:
  RfInputs<T> params;
  RF_params rf_params;
  T *data_d, *inference_data_d;
  std::vector<T> data_h;
  std::vector<T> inference_data_h;
  std::vector<ModelHandle> treelite_indiv_handles;

  // Set to 1 for regression and 2 for binary classification
  // #class for multi-classification
  int task_category;
  int is_classification;

  int data_len;
  int inference_data_len;

  hipStream_t stream;
  std::shared_ptr<raft::handle_t> handle;
  std::vector<float> treelite_predicted_labels;
  std::vector<float> ref_predicted_labels;
  std::vector<ML::RandomForestMetaData<T, L> *> all_forest_info;
  std::string test_dir;
  std::string dir_name;

  L *labels_d, *predicted_labels_d;
  std::vector<L> labels_h;
  std::vector<L> predicted_labels_h;

  RandomForestMetaData<T, L> *forest;
  RandomForestMetaData<T, L> *forest_2;
  RandomForestMetaData<T, L> *forest_3;
};  // namespace ML

template <typename T, typename L>
class RfConcatTestClf : public RfTreeliteTestCommon<T, L> {
 protected:
  void testClassifier() {
    this->test_dir = "./concat_test_clf/";
    this->is_classification = 1;
    //task_category - 1 for regression, 2 for binary classification
    // #class for multi-class classification
    this->task_category = 2;

    float *weight, *temp_label_d, *temp_data_d;
    std::vector<float> temp_label_h;

    raft::allocate(weight, this->params.n_cols);
    raft::allocate(temp_label_d, this->params.n_rows);
    raft::allocate(temp_data_d, this->data_len);

    raft::random::Rng r(1234ULL);

    // Generate weight for each feature.
    r.uniform(weight, this->params.n_cols, T(0.0), T(1.0), this->stream);
    // Generate noise.
    r.uniform(temp_label_d, this->params.n_rows, T(0.0), T(10.0), this->stream);

    raft::linalg::transpose<float>(*(this->handle), this->data_d, temp_data_d,
                                   this->params.n_rows, this->params.n_cols,
                                   this->stream);

    raft::linalg::gemv<float>(*(this->handle), temp_data_d, this->params.n_cols,
                              this->params.n_rows, weight, temp_label_d, true,
                              1.f, 1.f, this->stream);

    temp_label_h.resize(this->params.n_rows);
    raft::update_host(temp_label_h.data(), temp_label_d, this->params.n_rows,
                      this->stream);

    CUDA_CHECK(hipStreamSynchronize(this->stream));

    int value;
    for (int i = 0; i < this->params.n_rows; i++) {
      // The value of temp_label is between 0 to 10*n_cols+noise_level(10).
      // Choose half of that as the theshold to balance two classes.
      if (temp_label_h[i] >= (10 * this->params.n_cols + 10) / 2.0) {
        value = 1;
      } else {
        value = 0;
      }
      this->labels_h.push_back(value);
    }

    raft::update_device(this->labels_d, this->labels_h.data(),
                        this->params.n_rows, this->stream);

    preprocess_labels(this->params.n_rows, this->labels_h, labels_map);

    for (int i = 0; i < 3; i++) {
      ModelHandle model;

      this->rf_params.n_trees = this->rf_params.n_trees + i;

      fit(*(this->handle), this->all_forest_info[i], this->data_d,
          this->params.n_rows, this->params.n_cols, this->labels_d,
          labels_map.size(), this->rf_params);
      build_treelite_forest(&model, this->all_forest_info[i],
                            this->params.n_cols, this->task_category);
      this->treelite_indiv_handles.push_back(model);
    }

    CUDA_CHECK(hipStreamSynchronize(this->stream));

    this->ConcatenateTreeliteModels();
    this->getResultAndCheck();

    postprocess_labels(this->params.n_rows, this->labels_h, this->labels_map);

    labels_map.clear();
    temp_label_h.clear();
    CUDA_CHECK(hipFree(weight));
    CUDA_CHECK(hipFree(temp_label_d));
    CUDA_CHECK(hipFree(temp_data_d));
  }

 protected:
  std::map<int, int>
    labels_map;  //unique map of labels to int vals starting from 0
};

//-------------------------------------------------------------------------------------------------------------------------------------
template <typename T, typename L>
class RfConcatTestReg : public RfTreeliteTestCommon<T, L> {
 protected:
  void testRegressor() {
    this->test_dir = "./concat_test_reg/";
    this->is_classification = 0;
    // task_category - 1 for regression, 2 for binary classification
    // #class for multi-class classification
    this->task_category = 1;

    float *weight, *temp_data_d;
    raft::allocate(weight, this->params.n_cols);
    raft::allocate(temp_data_d, this->data_len);

    raft::random::Rng r(1234ULL);

    // Generate weight for each feature.
    r.uniform(weight, this->params.n_cols, T(0.0), T(1.0), this->stream);
    // Generate noise.
    r.uniform(this->labels_d, this->params.n_rows, T(0.0), T(10.0),
              this->stream);

    raft::linalg::transpose<float>(*(this->handle), this->data_d, temp_data_d,
                                   this->params.n_rows, this->params.n_cols,
                                   this->stream);

    raft::linalg::gemv<float>(*(this->handle), temp_data_d, this->params.n_cols,
                              this->params.n_rows, weight, this->labels_d, true,
                              1.f, 1.f, this->stream);

    this->labels_h.resize(this->params.n_rows);
    raft::update_host(this->labels_h.data(), this->labels_d,
                      this->params.n_rows, this->stream);
    CUDA_CHECK(hipStreamSynchronize(this->stream));

    for (int i = 0; i < 3; i++) {
      ModelHandle model;

      this->rf_params.n_trees = this->rf_params.n_trees + i;

      fit(*(this->handle), this->all_forest_info[i], this->data_d,
          this->params.n_rows, this->params.n_cols, this->labels_d,
          this->rf_params);
      build_treelite_forest(&model, this->all_forest_info[i],
                            this->params.n_cols, this->task_category);
      CUDA_CHECK(hipStreamSynchronize(this->stream));
      this->treelite_indiv_handles.push_back(model);
    }

    this->ConcatenateTreeliteModels();
    this->getResultAndCheck();

    CUDA_CHECK(hipFree(weight));
    CUDA_CHECK(hipFree(temp_data_d));
  }
};

// //-------------------------------------------------------------------------------------------------------------------------------------
const std::vector<RfInputs<float>> inputsf2_clf = {
  {4, 2, 1, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::GINI},  // single tree forest, bootstrap false, depth 8, 4 bins
  {4, 2, 1, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2,
   CRITERION::GINI},  // single tree forest, bootstrap false, depth of 8, 4 bins
  {4, 2, 10, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2,
   CRITERION::
     GINI},  //forest with 10 trees, all trees should produce identical predictions (no bootstrapping or column subsampling)
  {4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2,
   CRITERION::
     GINI},  //forest with 10 trees, with bootstrap and column subsampling enabled, 3 bins
  {4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::GLOBAL_QUANTILE,
   2, 2, 0.0, 2,
   CRITERION::
     CRITERION_END},  //forest with 10 trees, with bootstrap and column subsampling enabled, 3 bins, different split algorithm
  {4, 2, 1, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::ENTROPY},
  {4, 2, 1, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::ENTROPY},
  {4, 2, 10, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::ENTROPY},
  {4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::ENTROPY},
  {4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::GLOBAL_QUANTILE,
   2, 2, 0.0, 2, CRITERION::ENTROPY}};

typedef RfConcatTestClf<float, int> RfClassifierConcatTestF;
TEST_P(RfClassifierConcatTestF, Convert_Clf) { testClassifier(); }

INSTANTIATE_TEST_CASE_P(RfBinaryClassifierConcatTests, RfClassifierConcatTestF,
                        ::testing::ValuesIn(inputsf2_clf));

const std::vector<RfInputs<float>> inputsf2_reg = {
  {4, 2, 1, 1.0f, 1.0f, 4, 7, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::MSE},
  {4, 2, 1, 1.0f, 1.0f, 4, 7, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::MSE},
  {4, 2, 5, 1.0f, 1.0f, 4, 7, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2,
   CRITERION::
     CRITERION_END},  // CRITERION_END uses the default criterion (GINI for classification, MSE for regression)
  {4, 2, 1, 1.0f, 1.0f, 4, 7, -1, false, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::MAE},
  {4, 2, 1, 1.0f, 1.0f, 4, 7, -1, false, false, 4, SPLIT_ALGO::GLOBAL_QUANTILE,
   2, 2, 0.0, 2, CRITERION::MAE},
  {4, 2, 5, 1.0f, 1.0f, 4, 7, -1, true, false, 4, SPLIT_ALGO::HIST, 2, 2, 0.0,
   2, CRITERION::CRITERION_END}};

typedef RfConcatTestReg<float, float> RfRegressorConcatTestF;
TEST_P(RfRegressorConcatTestF, Convert_Reg) { testRegressor(); }

INSTANTIATE_TEST_CASE_P(RfRegressorConcatTests, RfRegressorConcatTestF,
                        ::testing::ValuesIn(inputsf2_reg));
}  // end namespace ML
