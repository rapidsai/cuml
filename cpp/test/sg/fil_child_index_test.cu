#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../../src/fil/internal.cuh"

#include <test_utils.h>

#include <cuml/fil/fil.h>
#include <gtest/gtest.h>

#include <cmath>
#include <cstdio>
#include <limits>
#include <memory>
#include <numeric>
#include <ostream>
#include <utility>

namespace ML {

using namespace fil;

struct proto_inner_node {
  bool def_left = false, is_categorical = false;
  int fid      = 0;     // feature id
  int set      = 0;     // which bit set represents the matching category list
  float thresh = 0.0f;  // threshold
  int left     = 1;     // left child idx
  val_t split()
  {
    val_t split;
    if (is_categorical)
      split.idx = set;
    else
      split.f = thresh;
    return split;
  }
  operator sparse_node16()
  {
    return sparse_node16({}, split(), fid, def_left, false, is_categorical, left);
  }
  operator sparse_node8()
  {
    return sparse_node8({}, split(), fid, def_left, false, is_categorical, left);
  }
  operator dense_node() { return dense_node({}, split(), fid, def_left, false, is_categorical); }
};

std::ostream& operator<<(std::ostream& os, const proto_inner_node& node)
{
  os << "def_left " << node.def_left << " is_categorical " << node.is_categorical << " fid "
     << node.fid << " set " << node.set << " thresh " << node.thresh << " left " << node.left;
  return os;
}

// proto inner node
#define NODE(...)                                               \
  []() {                                                        \
    struct NonDefaultProtoInnerNode : public proto_inner_node { \
      NonDefaultProtoInnerNode() { __VA_ARGS__; }               \
    };                                                          \
    return proto_inner_node(NonDefaultProtoInnerNode());        \
  }()

// proto category sets for one node
struct PCS {
  // each bit set for each feature id is in a separate vector
  // read each uint8_t from right to left, and the vector(s) - from left to right
  std::vector<std::vector<uint8_t>> bits;
  std::vector<int> max_matching;
  operator cat_sets_owner()
  {
    ASSERT(bits.size() == max_matching.size(),
           "internal error: PCS::bits.size() != PCS::max_matching.size()");
    std::vector<uint8_t> flat;
    for (std::vector<uint8_t> v : bits) {
      for (uint8_t b : v)
        flat.push_back(b);
    }
    return {flat, max_matching};
  }
};

struct ChildIdxTestParams {
  proto_inner_node node;
  int parent_node_idx = 0;
  cat_sets_owner cso;
  float input = 0.0f;
  int correct = INT_MAX;
};

std::ostream& operator<<(std::ostream& os, const ChildIdxTestParams& ps)
{
  os << "node = {\n"
     << ps.node << "\n} "
     << "parent_node_idx = " << ps.parent_node_idx << " cat_sets_owner = {\n"
     << ps.cso << "\n} input = " << ps.input << " correct = " << ps.correct;
  return os;
}

/** mechanism to use named aggregate initialization before C++20, and also use
    the struct defaults. Using it directly only works if all defaulted
    members come after ones explicitly mentioned.
**/
#define CHILD_IDX_TEST_PARAMS(...)                                    \
  []() {                                                              \
    struct NonDefaultChildIdxTestParams : public ChildIdxTestParams { \
      NonDefaultChildIdxTestParams() { __VA_ARGS__; }                 \
    };                                                                \
    return ChildIdxTestParams(NonDefaultChildIdxTestParams());        \
  }()

template <typename fil_node_t>
class ChildIdxTest : public testing::TestWithParam<ChildIdxTestParams> {
 protected:
  void check()
  {
    ChildIdxTestParams param = GetParam();
    tree_base tree{param.cso.accessor()};
    // nan -> !def_left, categorical -> if matches, numerical -> input >= threshold
    int test_idx =
      tree.child_index<true>((fil_node_t)param.node, param.parent_node_idx, param.input);
    ASSERT(test_idx == param.correct,
           "child index test: actual %d != correct %d",
           test_idx,
           param.correct);
  }
};

typedef ChildIdxTest<fil::dense_node> ChildIdxTestDense;

/* for dense nodes, left (false) == parent * 2 + 1, right (true) == parent * 2 + 2
   E.g. see tree below:
 0 -> 1, 2
 1 -> 3, 4
 2 -> 5, 6
 3 -> 7, 8
 4 -> 9, 10
 */
const float INF = std::numeric_limits<float>::infinity();

std::vector<ChildIdxTestParams> dense_params = {
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, input = -INF, correct = 1),   // val !>= thresh
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, input = 0.0f, correct = 2),   // val >= thresh
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, input = +INF, correct = 2),   // val >= thresh
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, input = NAN, correct = 2),    // !def_left
  CHILD_IDX_TEST_PARAMS(node.def_left = true, input = NAN, correct = 1),  // !def_left
  CHILD_IDX_TEST_PARAMS(node.thresh = NAN, input = NAN, correct = 2),     // !def_left
  CHILD_IDX_TEST_PARAMS(
    node.def_left = true, node.thresh = NAN, input = NAN, correct = 1),  // !def_left
  CHILD_IDX_TEST_PARAMS(node.thresh = NAN, input = 0.0f, correct = 1),   // val !>= thresh
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 1, input = -INF, correct = 3),
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 1, input = 0.0f, correct = 4),
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 2, input = -INF, correct = 5),
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 2, input = 0.0f, correct = 6),
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 3, input = -INF, correct = 7),
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 3, input = 0.0f, correct = 8),
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 4, input = -INF, correct = 9),
  CHILD_IDX_TEST_PARAMS(node.thresh = 0.0f, parent_node_idx = 4, input = 0.0f, correct = 10),
  CHILD_IDX_TEST_PARAMS(parent_node_idx = 4, input = NAN, correct = 10),  // !def_left
  CHILD_IDX_TEST_PARAMS(
    node.def_left = true, input = NAN, parent_node_idx = 4, correct = 9),  // !def_left
  // cannot match ( > max_matching)
  CHILD_IDX_TEST_PARAMS(
    node.is_categorical = true, cso.bits = {}, cso.max_matching = {-1}, input = 0, correct = 1),
  // doesn't match (bits[category] == 0, category == 0)
  CHILD_IDX_TEST_PARAMS(node.is_categorical = true,
                        cso.bits            = {0b0000'0000},
                        cso.max_matching    = {0},
                        input               = 0,
                        correct             = 1),
  // matches
  CHILD_IDX_TEST_PARAMS(node.is_categorical = true,
                        cso.bits            = {0b0000'0001},
                        cso.max_matching    = {0},
                        input               = 0,
                        correct             = 2),
  // matches
  CHILD_IDX_TEST_PARAMS(node.is_categorical = true,
                        cso.bits            = {0b0000'0101},
                        cso.max_matching    = {2, -1},
                        input               = 2,
                        correct             = 2),
  // doesn't match (bits[category] == 0, category > 0)
  CHILD_IDX_TEST_PARAMS(node.is_categorical = true,
                        cso.bits            = {0b0000'0101},
                        cso.max_matching    = {2},
                        input               = 1,
                        correct             = 1),
  // canot match (max_matching[fid=1] == -1)
  CHILD_IDX_TEST_PARAMS(node.is_categorical = true,
                        node.fid            = 1,
                        cso.bits            = {0b0000'0101},
                        cso.max_matching    = {2, -1},
                        input               = 2,
                        correct             = 1),
};

TEST_P(ChildIdxTestDense, Predict) { check(); }

INSTANTIATE_TEST_CASE_P(FilTests, ChildIdxTestDense, testing::ValuesIn(dense_params));
}  // namespace ML
