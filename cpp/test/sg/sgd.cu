/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/cusolver_wrappers.h>
#include <test_utils.h>
#include <raft/matrix/matrix.cuh>
#include <solver/sgd.cuh>

namespace ML {
namespace Solver {

using namespace MLCommon;

template <typename T>
struct SgdInputs {
  T tol;
  int n_row;
  int n_col;
  int n_row2;
  int n_col2;
  int batch_size;
};

template <typename T>
class SgdTest : public ::testing::TestWithParam<SgdInputs<T>> {
 protected:
  void linearRegressionTest() {
    params = ::testing::TestWithParam<SgdInputs<T>>::GetParam();
    int len = params.n_row * params.n_col;

    raft::allocate(data, len);
    raft::allocate(labels, params.n_row);
    raft::allocate(coef, params.n_col, true);
    raft::allocate(coef2, params.n_col, true);
    raft::allocate(coef_ref, params.n_col);
    raft::allocate(coef2_ref, params.n_col);

    T data_h[len] = {1.0, 1.0, 2.0, 2.0, 1.0, 2.0, 2.0, 3.0};
    raft::update_device(data, data_h, len, stream);

    T labels_h[params.n_row] = {6.0, 8.0, 9.0, 11.0};
    raft::update_device(labels, labels_h, params.n_row, stream);

    T coef_ref_h[params.n_col] = {2.087, 2.5454557};
    raft::update_device(coef_ref, coef_ref_h, params.n_col, stream);

    T coef2_ref_h[params.n_col] = {1.000001, 1.9999998};
    raft::update_device(coef2_ref, coef2_ref_h, params.n_col, stream);

    bool fit_intercept = false;
    intercept = T(0);
    int epochs = 2000;
    T lr = T(0.01);
    ML::lr_type lr_type = ML::lr_type::ADAPTIVE;
    T power_t = T(0.5);
    T alpha = T(0.0001);
    T l1_ratio = T(0.15);
    bool shuffle = true;
    T tol = T(1e-10);
    ML::loss_funct loss = ML::loss_funct::SQRD_LOSS;
    MLCommon::Functions::penalty pen = MLCommon::Functions::penalty::NONE;
    int n_iter_no_change = 10;

    sgdFit(handle, data, params.n_row, params.n_col, labels, coef, &intercept,
           fit_intercept, params.batch_size, epochs, lr_type, lr, power_t, loss,
           pen, alpha, l1_ratio, shuffle, tol, n_iter_no_change, stream);

    fit_intercept = true;
    intercept2 = T(0);
    sgdFit(handle, data, params.n_row, params.n_col, labels, coef2, &intercept2,
           fit_intercept, params.batch_size, epochs, ML::lr_type::CONSTANT, lr,
           power_t, loss, pen, alpha, l1_ratio, shuffle, tol, n_iter_no_change,
           stream);
  }

  void logisticRegressionTest() {
    params = ::testing::TestWithParam<SgdInputs<T>>::GetParam();
    int len = params.n_row2 * params.n_col2;

    T *coef_class;
    raft::allocate(data_logreg, len);
    raft::allocate(data_logreg_test, len);
    raft::allocate(labels_logreg, params.n_row2);
    raft::allocate(coef_class, params.n_col2, true);
    raft::allocate(pred_log, params.n_row2);
    raft::allocate(pred_log_ref, params.n_row2);

    T data_h[len] = {0.1,  -2.1, 5.4,  5.4,   -1.5,  -2.15,
                     2.65, 2.65, 3.25, -0.15, -7.35, -7.35};
    raft::update_device(data_logreg, data_h, len, stream);

    T data_test_h[len] = {0.3,   1.1,   2.1,  -10.1, 0.5,  2.5,
                          -3.55, -20.5, -1.3, 3.0,   -5.0, 15.0};
    raft::update_device(data_logreg_test, data_test_h, len, stream);

    T labels_logreg_h[params.n_row2] = {0.0, 1.0, 1.0, 0.0};
    raft::update_device(labels_logreg, labels_logreg_h, params.n_row2, stream);

    T pred_log_ref_h[params.n_row2] = {1.0, 0.0, 1.0, 1.0};
    raft::update_device(pred_log_ref, pred_log_ref_h, params.n_row2, stream);

    bool fit_intercept = true;
    T intercept_class = T(0);
    int epochs = 1000;
    T lr = T(0.05);
    ML::lr_type lr_type = ML::lr_type::CONSTANT;
    T power_t = T(0.5);
    T alpha = T(0.0);
    T l1_ratio = T(0.0);
    bool shuffle = false;
    T tol = T(0.0);
    ML::loss_funct loss = ML::loss_funct::LOG;
    MLCommon::Functions::penalty pen = MLCommon::Functions::penalty::NONE;
    int n_iter_no_change = 10;

    sgdFit(handle, data_logreg, params.n_row2, params.n_col2, labels_logreg,
           coef_class, &intercept_class, fit_intercept, params.batch_size,
           epochs, lr_type, lr, power_t, loss, pen, alpha, l1_ratio, shuffle,
           tol, n_iter_no_change, stream);

    sgdPredictBinaryClass(handle, data_logreg_test, params.n_row2,
                          params.n_col2, coef_class, intercept_class, pred_log,
                          loss, stream);

    CUDA_CHECK(hipFree(coef_class));
  }

  void svmTest() {
    params = ::testing::TestWithParam<SgdInputs<T>>::GetParam();
    int len = params.n_row2 * params.n_col2;

    T *coef_class;
    raft::allocate(data_svmreg, len);
    raft::allocate(data_svmreg_test, len);
    raft::allocate(labels_svmreg, params.n_row2);
    raft::allocate(coef_class, params.n_col2, true);
    raft::allocate(pred_svm, params.n_row2);
    raft::allocate(pred_svm_ref, params.n_row2);

    T data_h[len] = {0.1,  -2.1, 5.4,  5.4,   -1.5,  -2.15,
                     2.65, 2.65, 3.25, -0.15, -7.35, -7.35};
    raft::update_device(data_svmreg, data_h, len, stream);

    T data_test_h[len] = {0.3,   1.1,   2.1,  -10.1, 0.5,  2.5,
                          -3.55, -20.5, -1.3, 3.0,   -5.0, 15.0};
    raft::update_device(data_svmreg_test, data_test_h, len, stream);

    T labels_svmreg_h[params.n_row2] = {0.0, 1.0, 1.0, 0.0};
    raft::update_device(labels_svmreg, labels_svmreg_h, params.n_row2, stream);

    T pred_svm_ref_h[params.n_row2] = {1.0, 0.0, 1.0, 1.0};
    raft::update_device(pred_svm_ref, pred_svm_ref_h, params.n_row2, stream);

    bool fit_intercept = true;
    T intercept_class = T(0);
    int epochs = 1000;
    T lr = T(0.05);
    ML::lr_type lr_type = ML::lr_type::CONSTANT;
    T power_t = T(0.5);
    T alpha = T(1) / T(epochs);
    T l1_ratio = T(0.0);
    bool shuffle = false;
    T tol = T(0.0);
    ML::loss_funct loss = ML::loss_funct::HINGE;
    MLCommon::Functions::penalty pen = MLCommon::Functions::penalty::L2;
    int n_iter_no_change = 10;

    sgdFit(handle, data_svmreg, params.n_row2, params.n_col2, labels_svmreg,
           coef_class, &intercept_class, fit_intercept, params.batch_size,
           epochs, lr_type, lr, power_t, loss, pen, alpha, l1_ratio, shuffle,
           tol, n_iter_no_change, stream);

    sgdPredictBinaryClass(handle, data_svmreg_test, params.n_row2,
                          params.n_col2, coef_class, intercept_class, pred_svm,
                          loss, stream);

    CUDA_CHECK(hipFree(coef_class));
  }

  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);
    linearRegressionTest();
    logisticRegressionTest();
    svmTest();
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(coef));
    CUDA_CHECK(hipFree(coef_ref));
    CUDA_CHECK(hipFree(coef2));
    CUDA_CHECK(hipFree(coef2_ref));
    CUDA_CHECK(hipFree(data_logreg));
    CUDA_CHECK(hipFree(data_logreg_test));
    CUDA_CHECK(hipFree(labels_logreg));
    CUDA_CHECK(hipFree(data_svmreg));
    CUDA_CHECK(hipFree(data_svmreg_test));
    CUDA_CHECK(hipFree(labels_svmreg));
    CUDA_CHECK(hipFree(pred_svm));
    CUDA_CHECK(hipFree(pred_svm_ref));
    CUDA_CHECK(hipFree(pred_log));
    CUDA_CHECK(hipFree(pred_log_ref));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  SgdInputs<T> params;
  T *data, *labels, *coef, *coef_ref;
  T *coef2, *coef2_ref;
  T *data_logreg, *data_logreg_test, *labels_logreg;
  T *data_svmreg, *data_svmreg_test, *labels_svmreg;
  T *pred_svm, *pred_svm_ref, *pred_log, *pred_log_ref;
  T intercept, intercept2;
  hipStream_t stream;
  raft::handle_t handle;
};

const std::vector<SgdInputs<float>> inputsf2 = {{0.01f, 4, 2, 4, 3, 2}};

const std::vector<SgdInputs<double>> inputsd2 = {{0.01, 4, 2, 4, 3, 2}};

typedef SgdTest<float> SgdTestF;
TEST_P(SgdTestF, Fit) {
  ASSERT_TRUE(raft::devArrMatch(coef_ref, coef, params.n_col,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef2_ref, coef2, params.n_col,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred_log_ref, pred_log, params.n_row,
                                raft::CompareApproxAbs<float>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred_svm_ref, pred_svm, params.n_row,
                                raft::CompareApproxAbs<float>(params.tol)));
}

typedef SgdTest<double> SgdTestD;
TEST_P(SgdTestD, Fit) {
  ASSERT_TRUE(raft::devArrMatch(coef_ref, coef, params.n_col,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(coef2_ref, coef2, params.n_col,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred_log_ref, pred_log, params.n_row,
                                raft::CompareApproxAbs<double>(params.tol)));

  ASSERT_TRUE(raft::devArrMatch(pred_svm_ref, pred_svm, params.n_row,
                                raft::CompareApproxAbs<double>(params.tol)));
}

INSTANTIATE_TEST_CASE_P(SgdTests, SgdTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(SgdTests, SgdTestD, ::testing::ValuesIn(inputsd2));

}  // namespace Solver
}  // end namespace ML
