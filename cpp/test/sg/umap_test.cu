/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "cuML.hpp"

#include "knn/knn.hpp"
#include "umap/runner.h"
#include "umap/umapparams.h"

#include <cuda_utils.h>

#include <iostream>
#include <vector>

using namespace ML;
using namespace std;

/**
 * For now, this is mostly to test the c++ algorithm is able to be built.
 * Comprehensive comparisons of resulting embeddings are being done in the
 * Python test suite. Next to come will be a CUDA implementation of t-SNE's
 * trustworthiness score, which will allow us to gtest embedding algorithms.
 */
class UMAPTest : public ::testing::Test {
 protected:
  void basicTest() {
    cumlHandle handle;

    umap_params = new UMAPParams();
    umap_params->n_neighbors = k;
    umap_params->verbose = true;
    umap_params->target_metric = UMAPParams::MetricType::CATEGORICAL;

    kNN *knn = new kNN(handle, d);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    UMAPAlgo::find_ab(umap_params, stream);

    std::vector<float> X = {1.0,  1.0, 34.0, 76.0, 2.0, 29.0,
                            34.0, 3.0, 13.0, 23.0, 7.0, 80.0};

    std::vector<float> Y = {-1, 1, 1, 0};

    float *X_d, *Y_d;
    MLCommon::allocate(Y_d, n);
    MLCommon::allocate(X_d, n * d);
    MLCommon::updateDevice(X_d, X.data(), n * d, stream);
    MLCommon::updateDevice(Y_d, Y.data(), n, stream);

    MLCommon::allocate(embeddings, n * umap_params->n_components);

    std::cout << "Performing fit()" << std::endl;

    UMAPAlgo::_fit<float, 256>(handle, X_d, n, d, knn, umap_params, embeddings,
                               stream);

    std::cout << "done." << std::endl;

    std::cout << "Performing transform" << std::endl;

    float *xformed;
    MLCommon::allocate(xformed, n * umap_params->n_components);

    UMAPAlgo::_transform<float, 32>(handle, X_d, n, d, embeddings, n, knn,
                                    umap_params, xformed, stream);

    std::cout << "Done." << std::endl;

    std::cout << "Performing supervised fit" << std::endl;

    UMAPAlgo::_fit<float, 32>(handle, X_d, Y_d, n, d, knn, umap_params,
                              embeddings, stream);

    std::cout << "Done." << std::endl;

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {}

 protected:
  UMAPParams *umap_params;

  int d = 3;
  int n = 4;
  int k = 2;

  float *embeddings;
};

typedef UMAPTest UMAPTestF;
TEST_F(UMAPTestF, Result) {}
