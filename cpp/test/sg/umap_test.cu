/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "distance/distance.h"

#include "datasets/digits.h"

#include <cuml/manifold/umapparams.h>
#include <metrics/trustworthiness.h>
#include <cuml/common/cuml_allocator.hpp>
#include <cuml/cuml.hpp>
#include <cuml/neighbors/knn.hpp>

#include "common/device_buffer.hpp"
#include "umap/runner.h"

#include <cuda_utils.h>

#include <iostream>
#include <vector>

using namespace ML;
using namespace ML::Metrics;

using namespace std;

using namespace MLCommon;
using namespace MLCommon::Distance;
using namespace MLCommon::Datasets::Digits;

class UMAPTest : public ::testing::Test {
 protected:
  void xformTest() {
    cumlHandle handle;

    hipStream_t stream = handle.getStream();

    UMAPParams *umap_params = new UMAPParams();
    umap_params->n_neighbors = 10;
    umap_params->init = 1;
    umap_params->verbose = false;

    UMAPAlgo::find_ab(umap_params, handle.getDeviceAllocator(), stream);

    device_buffer<float> X_d(handle.getDeviceAllocator(), handle.getStream(),
                             n_samples * n_features);

    MLCommon::updateDevice(X_d.data(), digits.data(), n_samples * n_features,
                           handle.getStream());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    device_buffer<float> embeddings(handle.getDeviceAllocator(),
                                    handle.getStream(),
                                    n_samples * umap_params->n_components);

    UMAPAlgo::_fit<float, 256>(handle, X_d.data(), n_samples, n_features,
                               nullptr, nullptr, umap_params,
                               embeddings.data());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    device_buffer<float> xformed(handle.getDeviceAllocator(),
                                 handle.getStream(),
                                 n_samples * umap_params->n_components);

    UMAPAlgo::_transform<float, 256>(
      handle, X_d.data(), n_samples, n_features, nullptr, nullptr, X_d.data(),
      n_samples, embeddings.data(), n_samples, umap_params, xformed.data());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    xformed_score = trustworthiness_score<float, EucUnexpandedL2Sqrt>(
      handle, X_d.data(), xformed.data(), n_samples, n_features,
      umap_params->n_components, umap_params->n_neighbors);
  }

  void fitTest() {
    cumlHandle handle;

    hipStream_t stream = handle.getStream();

    UMAPParams *umap_params = new UMAPParams();
    umap_params->n_neighbors = 10;
    umap_params->init = 1;
    umap_params->verbose = false;

    UMAPAlgo::find_ab(umap_params, handle.getDeviceAllocator(), stream);

    device_buffer<float> X_d(handle.getDeviceAllocator(), handle.getStream(),
                             n_samples * n_features);

    MLCommon::updateDevice(X_d.data(), digits.data(), n_samples * n_features,
                           handle.getStream());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    device_buffer<float> embeddings(handle.getDeviceAllocator(),
                                    handle.getStream(),
                                    n_samples * umap_params->n_components);

    UMAPAlgo::_fit<float, 256>(handle, X_d.data(), n_samples, n_features,
                               nullptr, nullptr, umap_params,
                               embeddings.data());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    fit_score = trustworthiness_score<float, EucUnexpandedL2Sqrt>(
      handle, X_d.data(), embeddings.data(), n_samples, n_features,
      umap_params->n_components, umap_params->n_neighbors);
  }

  void supervisedTest() {
    cumlHandle handle;

    hipStream_t stream = handle.getStream();

    UMAPParams *umap_params = new UMAPParams();
    umap_params->n_neighbors = 10;
    umap_params->init = 1;
    umap_params->verbose = false;

    UMAPAlgo::find_ab(umap_params, handle.getDeviceAllocator(), stream);

    device_buffer<float> X_d(handle.getDeviceAllocator(), handle.getStream(),
                             n_samples * n_features);
    device_buffer<float> Y_d(handle.getDeviceAllocator(), handle.getStream(),
                             n_samples * 2);

    MLCommon::updateDevice(X_d.data(), digits.data(), n_samples * n_features,
                           handle.getStream());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    device_buffer<float> embeddings(handle.getDeviceAllocator(),
                                    handle.getStream(),
                                    n_samples * umap_params->n_components);

    UMAPAlgo::_fit<float, 256>(handle, X_d.data(), Y_d.data(), n_samples,
                               n_features, nullptr, nullptr, umap_params,
                               embeddings.data());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    supervised_score = trustworthiness_score<float, EucUnexpandedL2Sqrt>(
      handle, X_d.data(), embeddings.data(), n_samples, n_features,
      umap_params->n_components, umap_params->n_neighbors);
  }

  void fitWithKNNTest() {
    cumlHandle handle;

    UMAPParams *umap_params = new UMAPParams();
    umap_params->n_neighbors = 10;
    umap_params->init = 1;
    umap_params->verbose = false;

    UMAPAlgo::find_ab(umap_params, handle.getDeviceAllocator(),
                      handle.getStream());

    device_buffer<float> X_d(handle.getDeviceAllocator(), handle.getStream(),
                             n_samples * n_features);

    MLCommon::updateDevice(X_d.data(), digits.data(), n_samples * n_features,
                           handle.getStream());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    device_buffer<float> embeddings(handle.getDeviceAllocator(),
                                    handle.getStream(),
                                    n_samples * umap_params->n_components);

    MLCommon::device_buffer<int64_t> knn_indices(
      handle.getDeviceAllocator(), handle.getStream(),
      n_samples * umap_params->n_components);

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    MLCommon::device_buffer<float> knn_dists(
      handle.getDeviceAllocator(), handle.getStream(),
      n_samples * umap_params->n_components);

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    std::vector<float *> ptrs(1);
    std::vector<int> sizes(1);
    ptrs[0] = X_d.data();
    sizes[0] = n_samples;

    MLCommon::Selection::brute_force_knn(
      ptrs, sizes, n_features, X_d.data(), n_samples, knn_indices.data(),
      knn_dists.data(), umap_params->n_neighbors, handle.getDeviceAllocator(),
      handle.getStream());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    UMAPAlgo::_fit<float, 256>(
      handle, X_d.data(), n_samples, n_features,
      //knn_indices.data(), knn_dists.data(), umap_params,
      nullptr, nullptr, umap_params, embeddings.data());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    fit_with_knn_score = trustworthiness_score<float, EucUnexpandedL2Sqrt>(
      handle, X_d.data(), embeddings.data(), n_samples, n_features,
      umap_params->n_components, umap_params->n_neighbors);
  }

  void SetUp() override {
    fitTest();
    xformTest();
    supervisedTest();
    fitWithKNNTest();

    std::cout << "fit_score=" << fit_score << std::endl;
    std::cout << "xform_score=" << xformed_score << std::endl;
    std::cout << "supervised_score=" << supervised_score << std::endl;
    std::cout << "fit_with_knn_score=" << fit_with_knn_score << std::endl;
  }

  void TearDown() override {}

 protected:
  double fit_score;
  double xformed_score;
  double supervised_score;
  double fit_with_knn_score;
};

typedef UMAPTest UMAPTestF;
TEST_F(UMAPTestF, Result) {
  ASSERT_TRUE(fit_score > 0.98);
  ASSERT_TRUE(xformed_score > 0.80);
  ASSERT_TRUE(supervised_score > 0.98);
  ASSERT_TRUE(fit_with_knn_score > 0.96);
}
