/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "distance/distance.h"

#include "datasets/digits.h"

#include <cuml/manifold/umapparams.h>
#include <metrics/trustworthiness.h>
#include <cuml/common/cuml_allocator.hpp>
#include <cuml/cuml.hpp>
#include <cuml/neighbors/knn.hpp>

#include "common/device_buffer.hpp"
#include "umap/runner.h"

#include <cuda_utils.h>

#include <iostream>
#include <vector>

using namespace ML;
using namespace ML::Metrics;

using namespace std;

using namespace MLCommon;
using namespace MLCommon::Distance;
using namespace MLCommon::Datasets::Digits;

/**
 * For now, this is mostly to test the c++ algorithm is able to be built.
 * Comprehensive comparisons of resulting embeddings are being done in the
 * Python test suite. Next to come will be a CUDA implementation of t-SNE's
 * trustworthiness score, which will allow us to gtest embedding algorithms.
 */
class UMAPTest : public ::testing::Test {
 protected:
  void basicTest() {
    cumlHandle handle;

    hipStream_t stream = handle.getStream();

    umap_params = new UMAPParams();
    umap_params->n_neighbors = 15;
    umap_params->n_epochs = 500;
    umap_params->min_dist = 0.01;
    umap_params->verbose = false;

    UMAPAlgo::find_ab(umap_params, stream);

    /**
     * Allocate digits dataset
     */
    device_buffer<float> X_d(handle.getDeviceAllocator(), handle.getStream(),
                             n_samples * n_features);
    device_buffer<float> Y_d(handle.getDeviceAllocator(), handle.getStream(),
                             n_samples * 2);

    MLCommon::updateDevice(X_d.data(), digits.data(), n_samples * n_features,
                           handle.getStream());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    device_buffer<float> embeddings(handle.getDeviceAllocator(),
                                    handle.getStream(),
                                    n_samples * umap_params->n_components);

    UMAPAlgo::_fit<float, 32>(handle, X_d.data(), n_samples, n_features,
                              umap_params, embeddings.data());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    fit_score = trustworthiness_score<float, EucUnexpandedL2Sqrt>(
      handle, X_d.data(), embeddings.data(), n_samples, n_features,
      umap_params->n_components, umap_params->n_neighbors);

    device_buffer<float> xformed(handle.getDeviceAllocator(),
                                 handle.getStream(),
                                 n_samples * umap_params->n_components);

    UMAPAlgo::_transform<float, 32>(handle, X_d.data(), n_samples, n_features,
                                    X_d.data(), n_samples, embeddings.data(),
                                    n_samples, umap_params, xformed.data());

    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));

    xformed_score = trustworthiness_score<float, EucUnexpandedL2Sqrt>(
      handle, X_d.data(), xformed.data(), n_samples, n_features,
      umap_params->n_components, umap_params->n_neighbors);
    //
    //    UMAPAlgo::_fit<float, 32>(handle, X_d, Y_d, n, d, umap_params, embeddings);
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {}

 protected:
  UMAPParams *umap_params;

  double fit_score;
  double xformed_score;
  int k = 2;
};

typedef UMAPTest UMAPTestF;
TEST_F(UMAPTestF, Result) {
  ASSERT_TRUE(fit_score > 0.97);
  ASSERT_TRUE(xformed_score > 0.70);
}
