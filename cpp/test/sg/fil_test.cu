#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <test_utils.h>
#include <treelite/c_api.h>
#include <treelite/frontend.h>
#include <treelite/tree.h>
#include <cmath>
#include <cstdio>
#include <limits>
#include <memory>
#include <utility>
#include "cuml/fil/fil.h"
#include "ml_utils.h"
#include "random/rng.h"
#include "test_utils.h"

#define TL_CPP_CHECK(call) ASSERT(int(call) >= 0, "treelite call error")

namespace ML {

using namespace MLCommon;
namespace tl = treelite;
namespace tlf = treelite::frontend;

struct FilTestParams {
  // input data parameters
  int num_rows;
  int num_cols;
  float nan_prob;
  // forest parameters
  int depth;
  int num_trees;
  float leaf_prob;
  // output parameters
  fil::output_t output;
  float threshold;
  float global_bias;
  // runtime parameters
  fil::algo_t algo;
  int seed;
  float tolerance;
  // treelite parameters, only used for treelite tests
  tl::Operator op;
  fil::leaf_value_t leaf_payload_type;
  // num_classes must be 1 or 2 when FLOAT_SCALAR == leaf_payload_type
  // (1 if it's regression)
  // num_classes must be >1 when INT_CLASS_LABEL == leaf_payload_type
  // it's used in treelite ModelBuilder initialization
  int num_classes;

  size_t num_proba_outputs() { return num_rows * std::max(num_classes, 2); }
  size_t num_preds_outputs() { return num_rows; }
};

std::string output2str(fil::output_t output) {
  if (output == fil::RAW) return "RAW";
  std::string s = "";
  if (output & fil::AVG) s += "| AVG";
  if (output & fil::CLASS) s += "| CLASS";
  if (output & fil::SIGMOID) s += "| SIGMOID";
  return s;
}

std::ostream& operator<<(std::ostream& os, const FilTestParams& ps) {
  os << "num_rows = " << ps.num_rows << ", num_cols = " << ps.num_cols
     << ", nan_prob = " << ps.nan_prob << ", depth = " << ps.depth
     << ", num_trees = " << ps.num_trees << ", leaf_prob = " << ps.leaf_prob
     << ", output = " << output2str(ps.output)
     << ", threshold = " << ps.threshold << ", algo = " << ps.algo
     << ", seed = " << ps.seed << ", tolerance = " << ps.tolerance
     << ", op = " << tl::OpName(ps.op) << ", global_bias = " << ps.global_bias
     << ", leaf_payload_type = " << ps.leaf_payload_type
     << ", num_classes = " << ps.num_classes;
  return os;
}

__global__ void nan_kernel(float* data, const bool* mask, int len, float nan) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  if (!mask[tid]) data[tid] = nan;
}

float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

class BaseFilTest : public testing::TestWithParam<FilTestParams> {
 protected:
  void SetUp() override {
    // setup
    ps = testing::TestWithParam<FilTestParams>::GetParam();
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);

    generate_forest();
    generate_data();
    predict_on_cpu();
    predict_on_gpu();
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(preds_d));
    CUDA_CHECK(hipFree(want_preds_d));
    CUDA_CHECK(hipFree(data_d));
    CUDA_CHECK(hipFree(want_proba_d));
    CUDA_CHECK(hipFree(proba_d));
  }

  void generate_forest() {
    size_t num_nodes = forest_num_nodes();

    // helper data
    /// weights, used as float* or int*
    int* weights_d = nullptr;
    float* thresholds_d = nullptr;
    int* fids_d = nullptr;
    bool* def_lefts_d = nullptr;
    bool* is_leafs_d = nullptr;
    bool* def_lefts_h = nullptr;
    bool* is_leafs_h = nullptr;

    // allocate GPU data
    allocate(weights_d, num_nodes);
    // sizeof(float) == sizeof(int)
    allocate(thresholds_d, num_nodes);
    allocate(fids_d, num_nodes);
    allocate(def_lefts_d, num_nodes);
    allocate(is_leafs_d, num_nodes);

    // generate on-GPU random data
    Random::Rng r(ps.seed);
    if (ps.leaf_payload_type == fil::leaf_value_t::FLOAT_SCALAR) {
      r.uniform((float*)weights_d, num_nodes, -1.0f, 1.0f, stream);
    } else {
      // [0..num_classes)
      r.uniformInt((int*)weights_d, num_nodes, 0, ps.num_classes, stream);
    }
    r.uniform(thresholds_d, num_nodes, -1.0f, 1.0f, stream);
    r.uniformInt(fids_d, num_nodes, 0, ps.num_cols, stream);
    r.bernoulli(def_lefts_d, num_nodes, 0.5f, stream);
    r.bernoulli(is_leafs_d, num_nodes, 1.0f - ps.leaf_prob, stream);

    // copy data to host
    std::vector<float> thresholds_h(num_nodes);
    std::vector<int> weights_h(num_nodes), fids_h(num_nodes);
    def_lefts_h = new bool[num_nodes];
    is_leafs_h = new bool[num_nodes];

    updateHost(weights_h.data(), (int*)weights_d, num_nodes, stream);
    updateHost(thresholds_h.data(), thresholds_d, num_nodes, stream);
    updateHost(fids_h.data(), fids_d, num_nodes, stream);
    updateHost(def_lefts_h, def_lefts_d, num_nodes, stream);
    updateHost(is_leafs_h, is_leafs_d, num_nodes, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // mark leaves
    for (size_t i = 0; i < ps.num_trees; ++i) {
      int num_tree_nodes = tree_num_nodes();
      size_t leaf_start = num_tree_nodes * i + num_tree_nodes / 2;
      size_t leaf_end = num_tree_nodes * (i + 1);
      for (size_t j = leaf_start; j < leaf_end; ++j) {
        is_leafs_h[j] = true;
      }
    }

    // initialize nodes
    nodes.resize(num_nodes);
    for (size_t i = 0; i < num_nodes; ++i) {
      fil::val_t w;
      switch (ps.leaf_payload_type) {
        case fil::leaf_value_t::INT_CLASS_LABEL:
          w.idx = weights_h[i];
          break;
        case fil::leaf_value_t::FLOAT_SCALAR:
          // not relying on fil::val_t internals
          // merely that we copied floats into weights_h earlier
          std::memcpy(&w.f, &weights_h[i], sizeof w.f);
      }
      fil::dense_node_init(&nodes[i], w, thresholds_h[i], fids_h[i],
                           def_lefts_h[i], is_leafs_h[i]);
    }

    // clean up
    delete[] def_lefts_h;
    delete[] is_leafs_h;
    CUDA_CHECK(hipFree(is_leafs_d));
    CUDA_CHECK(hipFree(def_lefts_d));
    CUDA_CHECK(hipFree(fids_d));
    CUDA_CHECK(hipFree(thresholds_d));
    CUDA_CHECK(hipFree(weights_d));
  }

  void generate_data() {
    // allocate arrays
    size_t num_data = ps.num_rows * ps.num_cols;
    allocate(data_d, num_data);
    bool* mask_d = nullptr;
    allocate(mask_d, num_data);

    // generate random data
    Random::Rng r(ps.seed);
    r.uniform(data_d, num_data, -1.0f, 1.0f, stream);
    r.bernoulli(mask_d, num_data, ps.nan_prob, stream);
    int tpb = 256;
    nan_kernel<<<ceildiv(int(num_data), tpb), tpb, 0, stream>>>(
      data_d, mask_d, num_data, std::numeric_limits<float>::quiet_NaN());
    CUDA_CHECK(hipPeekAtLastError());

    // copy to host
    data_h.resize(num_data);
    updateHost(data_h.data(), data_d, num_data, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // clean up
    CUDA_CHECK(hipFree(mask_d));
  }

  void transform(float f, float& proba, float& output) {
    if ((ps.output & fil::output_t::AVG) != 0) {
      f *= (1.0f / ps.num_trees);
    }
    f += ps.global_bias;
    if ((ps.output & fil::output_t::SIGMOID) != 0) {
      f = sigmoid(f);
    }
    proba = f;
    if ((ps.output & fil::output_t::CLASS) != 0) {
      f = f > ps.threshold ? 1.0f : 0.0f;
    }
    output = f;
  }

  void complement(float* proba) { proba[0] = 1.0f - proba[1]; }

  void predict_on_cpu() {
    // predict on host
    std::vector<float> want_preds_h(ps.num_preds_outputs());
    std::vector<float> want_proba_h(ps.num_proba_outputs());
    int num_nodes = tree_num_nodes();
    switch (ps.leaf_payload_type) {
      case fil::leaf_value_t::FLOAT_SCALAR:
        for (int i = 0; i < ps.num_rows; ++i) {
          float pred = 0.0f;
          for (int j = 0; j < ps.num_trees; ++j) {
            pred +=
              infer_one_tree(&nodes[j * num_nodes], &data_h[i * ps.num_cols]).f;
          }
          transform(pred, want_proba_h[i * 2 + 1], want_preds_h[i]);
          complement(&(want_proba_h[i * 2]));
        }
        break;
      case fil::leaf_value_t::INT_CLASS_LABEL:
        std::vector<int> class_votes(ps.num_classes);
        for (int r = 0; r < ps.num_rows; ++r) {
          std::fill(class_votes.begin(), class_votes.end(), 0);
          for (int j = 0; j < ps.num_trees; ++j) {
            int class_label =
              infer_one_tree(&nodes[j * num_nodes], &data_h[r * ps.num_cols])
                .idx;
            ++class_votes[class_label];
          }
          for (int c = 0; c < ps.num_classes; ++c) {
            float thresholded_proba;  // not used; do argmax instead
            transform(class_votes[c], want_proba_h[r * ps.num_classes + c],
                      thresholded_proba);
          }
          want_preds_h[r] =
            std::max_element(class_votes.begin(), class_votes.end()) -
            class_votes.begin();
        }
        break;
    }

    // copy to GPU
    allocate(want_preds_d, ps.num_preds_outputs());
    allocate(want_proba_d, ps.num_proba_outputs());
    updateDevice(want_preds_d, want_preds_h.data(), ps.num_preds_outputs(),
                 stream);
    updateDevice(want_proba_d, want_proba_h.data(), ps.num_proba_outputs(),
                 stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  virtual void init_forest(fil::forest_t* pforest) = 0;

  void predict_on_gpu() {
    fil::forest_t forest = nullptr;
    init_forest(&forest);

    // predict
    allocate(preds_d, ps.num_preds_outputs());
    allocate(proba_d, ps.num_proba_outputs());
    fil::predict(handle, forest, preds_d, data_d, ps.num_rows);
    fil::predict(handle, forest, proba_d, data_d, ps.num_rows, true);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // cleanup
    fil::free(handle, forest);
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(want_proba_d, proba_d, ps.num_proba_outputs(),
                            CompareApprox<float>(ps.tolerance), stream));
    float tolerance = ps.leaf_payload_type == fil::leaf_value_t::FLOAT_SCALAR
                        ? ps.tolerance
                        : std::numeric_limits<float>::epsilon();
    // in multi-class prediction, floats represent the most likely class
    // and would be generated by converting an int to float
    ASSERT_TRUE(devArrMatch(want_preds_d, preds_d, ps.num_rows,
                            CompareApprox<float>(tolerance), stream));
  }

  fil::val_t infer_one_tree(fil::dense_node_t* root, float* data) {
    int curr = 0;
    float threshold = 0.0f;
    fil::val_t output{.f = 0.0f};
    int fid = 0;
    bool def_left = false, is_leaf = false;
    for (;;) {
      fil::dense_node_decode(&root[curr], &output, &threshold, &fid, &def_left,
                             &is_leaf);
      if (is_leaf) break;
      float val = data[fid];
      bool cond = isnan(val) ? !def_left : val >= threshold;
      curr = (curr << 1) + 1 + (cond ? 1 : 0);
    }
    return output;
  }

  int tree_num_nodes() { return (1 << (ps.depth + 1)) - 1; }

  int forest_num_nodes() { return tree_num_nodes() * ps.num_trees; }

  // predictions
  float* preds_d = nullptr;
  float* proba_d = nullptr;
  float* want_preds_d = nullptr;
  float* want_proba_d = nullptr;

  // input data
  float* data_d = nullptr;
  std::vector<float> data_h;

  // forest data
  std::vector<fil::dense_node_t> nodes;

  // parameters
  hipStream_t stream;
  cumlHandle handle;
  FilTestParams ps;
};

class PredictDenseFilTest : public BaseFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    // init FIL model
    fil::forest_params_t fil_ps;
    fil_ps.depth = ps.depth;
    fil_ps.num_trees = ps.num_trees;
    fil_ps.num_cols = ps.num_cols;
    fil_ps.algo = ps.algo;
    fil_ps.output = ps.output;
    fil_ps.threshold = ps.threshold;
    fil_ps.global_bias = ps.global_bias;
    fil_ps.leaf_payload_type = ps.leaf_payload_type;
    fil_ps.num_classes = ps.num_classes;
    fil::init_dense(handle, pforest, nodes.data(), &fil_ps);
  }
};

class PredictSparseFilTest : public BaseFilTest {
 protected:
  void dense2sparse_node(const fil::dense_node_t* dense_root, int i_dense,
                         int i_sparse_root, int i_sparse) {
    float threshold;
    fil::val_t output;
    int feature;
    bool def_left, is_leaf;
    dense_node_decode(&dense_root[i_dense], &output, &threshold, &feature,
                      &def_left, &is_leaf);
    if (is_leaf) {
      // leaf sparse node
      sparse_node_init(&sparse_nodes[i_sparse], output, threshold, feature,
                       def_left, is_leaf, 0);
      return;
    }
    // inner sparse node
    // reserve space for children
    int left_index = sparse_nodes.size();
    sparse_nodes.push_back(fil::sparse_node_t());
    sparse_nodes.push_back(fil::sparse_node_t());
    sparse_node_init(&sparse_nodes[i_sparse], output, threshold, feature,
                     def_left, is_leaf, left_index - i_sparse_root);
    dense2sparse_node(dense_root, 2 * i_dense + 1, i_sparse_root, left_index);
    dense2sparse_node(dense_root, 2 * i_dense + 2, i_sparse_root,
                      left_index + 1);
  }

  void dense2sparse_tree(const fil::dense_node_t* dense_root) {
    int i_sparse_root = sparse_nodes.size();
    sparse_nodes.push_back(fil::sparse_node_t());
    dense2sparse_node(dense_root, 0, i_sparse_root, i_sparse_root);
    trees.push_back(i_sparse_root);
  }

  void dense2sparse() {
    for (int tree = 0; tree < ps.num_trees; ++tree) {
      dense2sparse_tree(&nodes[tree * tree_num_nodes()]);
    }
  }

  void init_forest(fil::forest_t* pforest) override {
    // init FIL model
    fil::forest_params_t fil_params;
    fil_params.num_trees = ps.num_trees;
    fil_params.num_cols = ps.num_cols;
    fil_params.algo = ps.algo;
    fil_params.output = ps.output;
    fil_params.threshold = ps.threshold;
    fil_params.global_bias = ps.global_bias;
    fil_params.leaf_payload_type = ps.leaf_payload_type;
    fil_params.num_classes = ps.num_classes;
    dense2sparse();
    fil_params.num_nodes = sparse_nodes.size();
    fil::init_sparse(handle, pforest, trees.data(), sparse_nodes.data(),
                     &fil_params);
  }
  std::vector<fil::sparse_node_t> sparse_nodes;
  std::vector<int> trees;
};

class TreeliteFilTest : public BaseFilTest {
 protected:
  /** adds nodes[node] of tree starting at index root to builder
      at index at *pkey, increments *pkey,
      and returns the treelite key of the node */
  int node_to_treelite(tlf::TreeBuilder* builder, int* pkey, int root,
                       int node) {
    int key = (*pkey)++;
    TL_CPP_CHECK(builder->CreateNode(key));
    int feature;
    float threshold;
    fil::val_t output;
    bool is_leaf, default_left;
    fil::dense_node_decode(&nodes[node], &output, &threshold, &feature,
                           &default_left, &is_leaf);
    if (is_leaf) {
      switch (ps.leaf_payload_type) {
        case fil::leaf_value_t::FLOAT_SCALAR:
          // default is fil::FLOAT_SCALAR
          TL_CPP_CHECK(builder->SetLeafNode(key, output.f));
          break;
        case fil::leaf_value_t::INT_CLASS_LABEL:
          std::vector<tl::tl_float> vec(ps.num_classes);
          for (int i = 0; i < ps.num_classes; ++i)
            vec[i] = i == output.idx ? 1.0f : 0.0f;
          TL_CPP_CHECK(builder->SetLeafVectorNode(key, vec));
      }
    } else {
      int left = root + 2 * (node - root) + 1;
      int right = root + 2 * (node - root) + 2;
      switch (ps.op) {
        case tl::Operator::kLT:
          break;
        case tl::Operator::kLE:
          // adjust the threshold
          threshold =
            std::nextafterf(threshold, -std::numeric_limits<float>::infinity());
          break;
        case tl::Operator::kGT:
          // adjust the threshold; left and right still need to be swapped
          threshold =
            std::nextafterf(threshold, -std::numeric_limits<float>::infinity());
        case tl::Operator::kGE:
          // swap left and right
          std::swap(left, right);
          default_left = !default_left;
          break;
        default:
          ASSERT(false, "comparison operator must be <, >, <= or >=");
      }
      int left_key = node_to_treelite(builder, pkey, root, left);
      int right_key = node_to_treelite(builder, pkey, root, right);
      TL_CPP_CHECK(builder->SetNumericalTestNode(
        key, feature, ps.op, threshold, default_left, left_key, right_key));
    }
    return key;
  }

  void init_forest_impl(fil::forest_t* pforest,
                        fil::storage_type_t storage_type) {
    bool random_forest_flag = (ps.output & fil::output_t::AVG) != 0;
    int treelite_num_classes =
      ps.leaf_payload_type == fil::leaf_value_t::FLOAT_SCALAR ? 1
                                                              : ps.num_classes;
    std::unique_ptr<tlf::ModelBuilder> model_builder(new tlf::ModelBuilder(
      ps.num_cols, treelite_num_classes, random_forest_flag));

    // prediction transform
    if ((ps.output & fil::output_t::SIGMOID) != 0) {
      model_builder->SetModelParam("pred_transform", "sigmoid");
    }

    // global bias
    char* global_bias_str = nullptr;
    ASSERT(asprintf(&global_bias_str, "%f", double(ps.global_bias)) > 0,
           "cannot convert global_bias into a string");
    model_builder->SetModelParam("global_bias", global_bias_str);
    free(global_bias_str);

    // build the trees
    for (int i_tree = 0; i_tree < ps.num_trees; ++i_tree) {
      tlf::TreeBuilder* tree_builder = new tlf::TreeBuilder();
      int key_counter = 0;
      int root = i_tree * tree_num_nodes();
      int root_key = node_to_treelite(tree_builder, &key_counter, root, root);
      TL_CPP_CHECK(tree_builder->SetRootNode(root_key));
      // InsertTree() consumes tree_builder
      TL_CPP_CHECK(model_builder->InsertTree(tree_builder));
    }

    // commit the model
    std::unique_ptr<tl::Model> model(new tl::Model);
    TL_CPP_CHECK(model_builder->CommitModel(model.get()));

    // init FIL forest with the model
    fil::treelite_params_t params;
    params.algo = ps.algo;
    params.threshold = ps.threshold;
    params.output_class = (ps.output & fil::output_t::CLASS) != 0;
    params.storage_type = storage_type;
    fil::from_treelite(handle, pforest, (ModelHandle)model.get(), &params);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
};

class TreeliteDenseFilTest : public TreeliteFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    init_forest_impl(pforest, fil::storage_type_t::DENSE);
  }
};

class TreeliteSparseFilTest : public TreeliteFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    init_forest_impl(pforest, fil::storage_type_t::SPARSE);
  }
};

class TreeliteAutoFilTest : public TreeliteFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    init_forest_impl(pforest, fil::storage_type_t::AUTO);
  }
};

// rows, cols, nan_prob, depth, num_trees, leaf_prob, output, threshold,
// global_bias, algo, seed, tolerance, branch comparison operator, FIL implementation, number of classes
std::vector<FilTestParams> predict_dense_inputs = {
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0.5,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0.5,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 1.0, 0.5,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 5},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 2},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 5},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 7},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0.5,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 4},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::INT_CLASS_LABEL, 4},
};

TEST_P(PredictDenseFilTest, Predict) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, PredictDenseFilTest,
                        testing::ValuesIn(predict_dense_inputs));

// rows, cols, nan_prob, depth, num_trees, leaf_prob, output, threshold,
// global_bias, algo, seed, tolerance, branch comparison operator, FIL implementation, number of classes
std::vector<FilTestParams> predict_sparse_inputs = {
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 1.0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 1.0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 5000},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::INT_CLASS_LABEL, 6},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 2},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator(0),
   fil::leaf_value_t::INT_CLASS_LABEL, 3},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator(0), fil::leaf_value_t::INT_CLASS_LABEL, 3},
};

TEST_P(PredictSparseFilTest, Predict) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, PredictSparseFilTest,
                        testing::ValuesIn(predict_sparse_inputs));

// rows, cols, nan_prob, depth, num_trees, leaf_prob, output, threshold,
// global_bias, algo, seed, tolerance, branch comparison operator, FIL implementation, number of classes
std::vector<FilTestParams> import_dense_inputs = {
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kLT, fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kGT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kGE, fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kGT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0.5,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0.5,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kGT, fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 1.0, 0.5,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::AVG), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::INT_CLASS_LABEL, 5},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kGT,
   fil::leaf_value_t::INT_CLASS_LABEL, 6},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::AVG), 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::INT_CLASS_LABEL, 3},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::INT_CLASS_LABEL, 5},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::INT_CLASS_LABEL, 5},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::TREE_REORG, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::INT_CLASS_LABEL, 7},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::INT_CLASS_LABEL, 2},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kLT, fil::leaf_value_t::INT_CLASS_LABEL, 6},
};

TEST_P(TreeliteDenseFilTest, Import) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteDenseFilTest,
                        testing::ValuesIn(import_dense_inputs));

// rows, cols, nan_prob, depth, num_trees, leaf_prob, output, threshold,
// global_bias, algo, seed, tolerance, branch comparison operator, FIL implementation, number of classes
std::vector<FilTestParams> import_sparse_inputs = {
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kLT, fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kGT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kGE, fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kLT, fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::SIGMOID, 0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kGT, fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 1.0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 1.0, 0.5,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kGE,
   fil::leaf_value_t::INT_CLASS_LABEL, 10},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::INT_CLASS_LABEL, 4},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::SIGMOID | fil::output_t::AVG), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLE,
   fil::leaf_value_t::INT_CLASS_LABEL, 5},
  {20000, 50, 0.05, 8, 50, 0.05,
   fil::output_t(fil::output_t::AVG | fil::output_t::CLASS), 0, 0,
   fil::algo_t::NAIVE, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::INT_CLASS_LABEL, 2},
  {20000, 50, 0.05, 8, 50, 0.05, fil::output_t::AVG, 0, 0.5, fil::algo_t::NAIVE,
   42, 2e-3f, tl::Operator::kLT, fil::leaf_value_t::INT_CLASS_LABEL, 3},
};

TEST_P(TreeliteSparseFilTest, Import) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteSparseFilTest,
                        testing::ValuesIn(import_sparse_inputs));

// rows, cols, nan_prob, depth, num_trees, leaf_prob, output, threshold,
// global_bias, algo, seed, tolerance, branch comparison operator, FIL implementation, number of classes
std::vector<FilTestParams> import_auto_inputs = {
  {20000, 50, 0.05, 10, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 15, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 19, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 19, 50, 0.05, fil::output_t::RAW, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::FLOAT_SCALAR, 0},
  {20000, 50, 0.05, 10, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::ALGO_AUTO, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::INT_CLASS_LABEL, 3},
  {20000, 50, 0.05, 19, 50, 0.05, fil::output_t::AVG, 0, 0,
   fil::algo_t::BATCH_TREE_REORG, 42, 2e-3f, tl::Operator::kLT,
   fil::leaf_value_t::INT_CLASS_LABEL, 6},
};

TEST_P(TreeliteAutoFilTest, Import) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteAutoFilTest,
                        testing::ValuesIn(import_auto_inputs));

}  // namespace ML
