#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/fil/fil.h>
#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <treelite/c_api.h>
#include <treelite/frontend.h>
#include <treelite/tree.h>
#include <cmath>
#include <cstdio>
#include <limits>
#include <memory>
#include <raft/cuda_utils.cuh>
#include <raft/random/rng.cuh>
#include <utility>

#include "../../src/fil/internal.cuh"

#define TL_CPP_CHECK(call) ASSERT(int(call) >= 0, "treelite call error")

namespace ML {

using namespace MLCommon;
namespace tl = treelite;
namespace tlf = treelite::frontend;
using namespace fil;

struct FilTestParams {
  // input data parameters
  int num_rows = 20'000;
  int num_cols = 50;
  float nan_prob = 0.05;
  // forest parameters
  int depth = 8;
  int num_trees = 50;
  float leaf_prob = 0.05;
  // output parameters
  output_t output = output_t::RAW;
  float threshold = 0.0f;
  float global_bias = 0.0f;
  // runtime parameters
  int blocks_per_sm = 0;
  algo_t algo = algo_t::NAIVE;
  int seed = 42;
  float tolerance = 2e-3f;
  // treelite parameters, only used for treelite tests
  tl::Operator op = tl::Operator::kLT;
  leaf_algo_t leaf_algo = leaf_algo_t::FLOAT_UNARY_BINARY;
  // when FLOAT_UNARY_BINARY == leaf_algo:
  // num_classes = 1 means it's regression
  // num_classes = 2 means it's binary classification
  // (complement probabilities, then use threshold)
  // when GROVE_PER_CLASS == leaf_algo:
  // it's multiclass classification (num_classes must be > 2),
  // done by splitting the forest in num_classes groups,
  // each of which computes one-vs-all probability for its class.
  // when CATEGORICAL_LEAF == leaf_algo:
  // num_classes must be > 1 and it's multiclass classification.
  // done by storing the class label in each leaf and voting.
  // it's used in treelite ModelBuilder initialization
  int num_classes = 1;

  size_t num_proba_outputs() { return num_rows * std::max(num_classes, 2); }
  size_t num_preds_outputs() { return num_rows; }
};

std::string output2str(fil::output_t output) {
  if (output == fil::RAW) return "RAW";
  std::string s = "";
  if (output & fil::AVG) s += "| AVG";
  if (output & fil::CLASS) s += "| CLASS";
  if (output & fil::SIGMOID) s += "| SIGMOID";
  if (output & fil::SOFTMAX) s += "| SOFTMAX";
  return s;
}

std::ostream& operator<<(std::ostream& os, const FilTestParams& ps) {
  os << "num_rows = " << ps.num_rows << ", num_cols = " << ps.num_cols
     << ", nan_prob = " << ps.nan_prob << ", depth = " << ps.depth
     << ", num_trees = " << ps.num_trees << ", leaf_prob = " << ps.leaf_prob
     << ", output = " << output2str(ps.output)
     << ", threshold = " << ps.threshold
     << ", blocks_per_sm = " << ps.blocks_per_sm << ", algo = " << ps.algo
     << ", seed = " << ps.seed << ", tolerance = " << ps.tolerance
     << ", op = " << tl::OpName(ps.op) << ", global_bias = " << ps.global_bias
     << ", leaf_algo = " << ps.leaf_algo
     << ", num_classes = " << ps.num_classes;
  return os;
}

__global__ void nan_kernel(float* data, const bool* mask, int len, float nan) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  if (!mask[tid]) data[tid] = nan;
}

float sigmoid(float x) { return 1.0f / (1.0f + expf(-x)); }

class BaseFilTest : public testing::TestWithParam<FilTestParams> {
 protected:
  void setup_helper() {
    // setup
    ps = testing::TestWithParam<FilTestParams>::GetParam();
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);

    generate_forest();
    generate_data();
    predict_on_cpu();
    predict_on_gpu();
  }

  void SetUp() override { setup_helper(); }

  void TearDown() override {
    CUDA_CHECK(hipFree(preds_d));
    CUDA_CHECK(hipFree(want_preds_d));
    CUDA_CHECK(hipFree(data_d));
    CUDA_CHECK(hipFree(want_proba_d));
    CUDA_CHECK(hipFree(proba_d));
  }

  void generate_forest() {
    size_t num_nodes = forest_num_nodes();

    // helper data
    /// weights, used as float* or int*
    int* weights_d = nullptr;
    float* thresholds_d = nullptr;
    int* fids_d = nullptr;
    bool* def_lefts_d = nullptr;
    bool* is_leafs_d = nullptr;
    bool* def_lefts_h = nullptr;
    bool* is_leafs_h = nullptr;

    // allocate GPU data
    raft::allocate(weights_d, num_nodes);
    // sizeof(float) == sizeof(int)
    raft::allocate(thresholds_d, num_nodes);
    raft::allocate(fids_d, num_nodes);
    raft::allocate(def_lefts_d, num_nodes);
    raft::allocate(is_leafs_d, num_nodes);

    // generate on-GPU random data
    raft::random::Rng r(ps.seed);
    if (ps.leaf_algo != fil::leaf_algo_t::CATEGORICAL_LEAF) {
      r.uniform((float*)weights_d, num_nodes, -1.0f, 1.0f, stream);
    } else {
      // [0..num_classes)
      r.uniformInt((int*)weights_d, num_nodes, 0, ps.num_classes, stream);
    }
    r.uniform(thresholds_d, num_nodes, -1.0f, 1.0f, stream);
    r.uniformInt(fids_d, num_nodes, 0, ps.num_cols, stream);
    r.bernoulli(def_lefts_d, num_nodes, 0.5f, stream);
    r.bernoulli(is_leafs_d, num_nodes, 1.0f - ps.leaf_prob, stream);

    // copy data to host
    std::vector<float> thresholds_h(num_nodes);
    std::vector<int> weights_h(num_nodes), fids_h(num_nodes);
    def_lefts_h = new bool[num_nodes];
    is_leafs_h = new bool[num_nodes];

    raft::update_host(weights_h.data(), (int*)weights_d, num_nodes, stream);
    raft::update_host(thresholds_h.data(), thresholds_d, num_nodes, stream);
    raft::update_host(fids_h.data(), fids_d, num_nodes, stream);
    raft::update_host(def_lefts_h, def_lefts_d, num_nodes, stream);
    raft::update_host(is_leafs_h, is_leafs_d, num_nodes, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // mark leaves
    for (size_t i = 0; i < ps.num_trees; ++i) {
      int num_tree_nodes = tree_num_nodes();
      size_t leaf_start = num_tree_nodes * i + num_tree_nodes / 2;
      size_t leaf_end = num_tree_nodes * (i + 1);
      for (size_t j = leaf_start; j < leaf_end; ++j) {
        is_leafs_h[j] = true;
      }
    }

    // initialize nodes
    nodes.resize(num_nodes);
    for (size_t i = 0; i < num_nodes; ++i) {
      fil::val_t w;
      switch (ps.leaf_algo) {
        case fil::leaf_algo_t::CATEGORICAL_LEAF:
          w.idx = weights_h[i];
          break;
        case fil::leaf_algo_t::FLOAT_UNARY_BINARY:
        case fil::leaf_algo_t::GROVE_PER_CLASS:
          // not relying on fil::val_t internals
          // merely that we copied floats into weights_h earlier
          std::memcpy(&w.f, &weights_h[i], sizeof w.f);
          break;
        default:
          ASSERT(false, "internal error: invalid ps.leaf_algo");
      }
      nodes[i] = fil::dense_node(w, thresholds_h[i], fids_h[i], def_lefts_h[i],
                                 is_leafs_h[i]);
    }

    // clean up
    delete[] def_lefts_h;
    delete[] is_leafs_h;
    CUDA_CHECK(hipFree(is_leafs_d));
    CUDA_CHECK(hipFree(def_lefts_d));
    CUDA_CHECK(hipFree(fids_d));
    CUDA_CHECK(hipFree(thresholds_d));
    CUDA_CHECK(hipFree(weights_d));
  }

  void generate_data() {
    // allocate arrays
    size_t num_data = ps.num_rows * ps.num_cols;
    raft::allocate(data_d, num_data);
    bool* mask_d = nullptr;
    raft::allocate(mask_d, num_data);

    // generate random data
    raft::random::Rng r(ps.seed);
    r.uniform(data_d, num_data, -1.0f, 1.0f, stream);
    r.bernoulli(mask_d, num_data, ps.nan_prob, stream);
    int tpb = 256;
    nan_kernel<<<raft::ceildiv(int(num_data), tpb), tpb, 0, stream>>>(
      data_d, mask_d, num_data, std::numeric_limits<float>::quiet_NaN());
    CUDA_CHECK(hipPeekAtLastError());

    // copy to host
    data_h.resize(num_data);
    raft::update_host(data_h.data(), data_d, num_data, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // clean up
    CUDA_CHECK(hipFree(mask_d));
  }

  void apply_softmax(float* class_scores) {
    float max = *std::max_element(class_scores, &class_scores[ps.num_classes]);
    for (int i = 0; i < ps.num_classes; ++i)
      class_scores[i] = expf(class_scores[i] - max);
    float sum =
      std::accumulate(class_scores, &class_scores[ps.num_classes], 0.0f);
    for (int i = 0; i < ps.num_classes; ++i) class_scores[i] /= sum;
  }

  void transform(float f, float& proba, float& output) {
    if ((ps.output & fil::output_t::AVG) != 0) {
      if (ps.leaf_algo == fil::leaf_algo_t::GROVE_PER_CLASS) {
        f /= ps.num_trees / ps.num_classes;
      } else {
        f *= 1.0f / ps.num_trees;
      }
    }
    f += ps.global_bias;
    if ((ps.output & fil::output_t::SIGMOID) != 0) {
      f = sigmoid(f);
    }
    proba = f;
    if ((ps.output & fil::output_t::CLASS) != 0) {
      f = f > ps.threshold ? 1.0f : 0.0f;
    }
    output = f;
  }

  void complement(float* proba) { proba[0] = 1.0f - proba[1]; }

  void predict_on_cpu() {
    // predict on host
    std::vector<float> want_preds_h(ps.num_preds_outputs());
    std::vector<float> want_proba_h(ps.num_proba_outputs());
    int num_nodes = tree_num_nodes();
    std::vector<float> class_scores(ps.num_classes);
    switch (ps.leaf_algo) {
      case fil::leaf_algo_t::FLOAT_UNARY_BINARY:
        for (int i = 0; i < ps.num_rows; ++i) {
          float pred = 0.0f;
          for (int j = 0; j < ps.num_trees; ++j) {
            pred +=
              infer_one_tree(&nodes[j * num_nodes], &data_h[i * ps.num_cols]).f;
          }
          transform(pred, want_proba_h[i * 2 + 1], want_preds_h[i]);
          complement(&(want_proba_h[i * 2]));
        }
        break;
      case fil::leaf_algo_t::GROVE_PER_CLASS:
        for (int row = 0; row < ps.num_rows; ++row) {
          std::fill(class_scores.begin(), class_scores.end(), 0.0f);
          for (int tree = 0; tree < ps.num_trees; ++tree) {
            class_scores[tree % ps.num_classes] +=
              infer_one_tree(&nodes[tree * num_nodes],
                             &data_h[row * ps.num_cols])
                .f;
          }
          want_preds_h[row] =
            std::max_element(class_scores.begin(), class_scores.end()) -
            class_scores.begin();
          for (int c = 0; c < ps.num_classes; ++c) {
            float thresholded_proba;  // not used;
            transform(class_scores[c], want_proba_h[row * ps.num_classes + c],
                      thresholded_proba);
          }
          if ((ps.output & fil::output_t::SOFTMAX) != 0)
            apply_softmax(&want_proba_h[row * ps.num_classes]);
        }
        break;
      case fil::leaf_algo_t::CATEGORICAL_LEAF:
        std::vector<int> class_votes(ps.num_classes);
        for (int r = 0; r < ps.num_rows; ++r) {
          std::fill(class_votes.begin(), class_votes.end(), 0);
          for (int j = 0; j < ps.num_trees; ++j) {
            int class_label =
              infer_one_tree(&nodes[j * num_nodes], &data_h[r * ps.num_cols])
                .idx;
            ++class_votes[class_label];
          }
          for (int c = 0; c < ps.num_classes; ++c) {
            float thresholded_proba;  // not used; do argmax instead
            transform(class_votes[c], want_proba_h[r * ps.num_classes + c],
                      thresholded_proba);
          }
          want_preds_h[r] =
            std::max_element(class_votes.begin(), class_votes.end()) -
            class_votes.begin();
        }
        break;
    }

    // copy to GPU
    raft::allocate(want_preds_d, ps.num_preds_outputs());
    raft::allocate(want_proba_d, ps.num_proba_outputs());
    raft::update_device(want_preds_d, want_preds_h.data(),
                        ps.num_preds_outputs(), stream);
    raft::update_device(want_proba_d, want_proba_h.data(),
                        ps.num_proba_outputs(), stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  virtual void init_forest(fil::forest_t* pforest) = 0;

  void predict_on_gpu() {
    fil::forest_t forest = nullptr;
    init_forest(&forest);

    // predict
    raft::allocate(preds_d, ps.num_preds_outputs());
    raft::allocate(proba_d, ps.num_proba_outputs());
    fil::predict(handle, forest, preds_d, data_d, ps.num_rows);
    fil::predict(handle, forest, proba_d, data_d, ps.num_rows, true);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // cleanup
    fil::free(handle, forest);
  }

  void compare() {
    ASSERT_TRUE(raft::devArrMatch(want_proba_d, proba_d, ps.num_proba_outputs(),
                                  raft::CompareApprox<float>(ps.tolerance),
                                  stream));
    float tolerance = ps.leaf_algo == fil::leaf_algo_t::FLOAT_UNARY_BINARY
                        ? ps.tolerance
                        : std::numeric_limits<float>::epsilon();
    // in multi-class prediction, floats represent the most likely class
    // and would be generated by converting an int to float
    ASSERT_TRUE(raft::devArrMatch(want_preds_d, preds_d, ps.num_rows,
                                  raft::CompareApprox<float>(tolerance),
                                  stream));
  }

  fil::val_t infer_one_tree(fil::dense_node* root, float* data) {
    int curr = 0;
    fil::val_t output{.f = 0.0f};
    for (;;) {
      const fil::dense_node& node = root[curr];
      if (node.is_leaf()) return node.base_node::output<val_t>();
      float val = data[node.fid()];
      bool cond = isnan(val) ? !node.def_left() : val >= node.thresh();
      curr = (curr << 1) + 1 + (cond ? 1 : 0);
    }
    return output;
  }

  int tree_num_nodes() { return (1 << (ps.depth + 1)) - 1; }

  int forest_num_nodes() { return tree_num_nodes() * ps.num_trees; }

  // predictions
  float* preds_d = nullptr;
  float* proba_d = nullptr;
  float* want_preds_d = nullptr;
  float* want_proba_d = nullptr;

  // input data
  float* data_d = nullptr;
  std::vector<float> data_h;

  // forest data
  std::vector<fil::dense_node> nodes;

  // parameters
  hipStream_t stream;
  raft::handle_t handle;
  FilTestParams ps;
};

class PredictDenseFilTest : public BaseFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    // init FIL model
    fil::forest_params_t fil_ps;
    fil_ps.depth = ps.depth;
    fil_ps.num_trees = ps.num_trees;
    fil_ps.num_cols = ps.num_cols;
    fil_ps.algo = ps.algo;
    fil_ps.output = ps.output;
    fil_ps.threshold = ps.threshold;
    fil_ps.global_bias = ps.global_bias;
    fil_ps.leaf_algo = ps.leaf_algo;
    fil_ps.num_classes = ps.num_classes;
    fil_ps.blocks_per_sm = ps.blocks_per_sm;

    fil::init_dense(handle, pforest, nodes.data(), &fil_ps);
  }
};

template <typename fil_node_t>
class BasePredictSparseFilTest : public BaseFilTest {
 protected:
  void dense2sparse_node(const fil::dense_node* dense_root, int i_dense,
                         int i_sparse_root, int i_sparse) {
    const fil::dense_node& node = dense_root[i_dense];
    if (node.is_leaf()) {
      // leaf sparse node
      sparse_nodes[i_sparse] =
        fil_node_t(node.base_node::output<val_t>(), node.thresh(), node.fid(),
                   node.def_left(), node.is_leaf(), 0);
      return;
    }
    // inner sparse node
    // reserve space for children
    int left_index = sparse_nodes.size();
    sparse_nodes.push_back(fil_node_t());
    sparse_nodes.push_back(fil_node_t());
    sparse_nodes[i_sparse] =
      fil_node_t(node.base_node::output<val_t>(), node.thresh(), node.fid(),
                 node.def_left(), node.is_leaf(), left_index - i_sparse_root);
    dense2sparse_node(dense_root, 2 * i_dense + 1, i_sparse_root, left_index);
    dense2sparse_node(dense_root, 2 * i_dense + 2, i_sparse_root,
                      left_index + 1);
  }

  void dense2sparse_tree(const fil::dense_node* dense_root) {
    int i_sparse_root = sparse_nodes.size();
    sparse_nodes.push_back(fil_node_t());
    dense2sparse_node(dense_root, 0, i_sparse_root, i_sparse_root);
    trees.push_back(i_sparse_root);
  }

  void dense2sparse() {
    for (int tree = 0; tree < ps.num_trees; ++tree) {
      dense2sparse_tree(&nodes[tree * tree_num_nodes()]);
    }
  }

  void init_forest(fil::forest_t* pforest) override {
    // init FIL model
    fil::forest_params_t fil_params;
    fil_params.num_trees = ps.num_trees;
    fil_params.num_cols = ps.num_cols;
    fil_params.algo = ps.algo;
    fil_params.output = ps.output;
    fil_params.threshold = ps.threshold;
    fil_params.global_bias = ps.global_bias;
    fil_params.leaf_algo = ps.leaf_algo;
    fil_params.num_classes = ps.num_classes;
    fil_params.blocks_per_sm = ps.blocks_per_sm;

    dense2sparse();
    fil_params.num_nodes = sparse_nodes.size();
    fil::init_sparse(handle, pforest, trees.data(), sparse_nodes.data(),
                     &fil_params);
  }
  std::vector<fil_node_t> sparse_nodes;
  std::vector<int> trees;
};

typedef BasePredictSparseFilTest<fil::sparse_node16> PredictSparse16FilTest;
typedef BasePredictSparseFilTest<fil::sparse_node8> PredictSparse8FilTest;

class TreeliteFilTest : public BaseFilTest {
 protected:
  /** adds nodes[node] of tree starting at index root to builder
      at index at *pkey, increments *pkey,
      and returns the treelite key of the node */
  int node_to_treelite(tlf::TreeBuilder* builder, int* pkey, int root,
                       int node) {
    int key = (*pkey)++;
    builder->CreateNode(key);
    const fil::dense_node& dense_node = nodes[node];
    if (dense_node.is_leaf()) {
      switch (ps.leaf_algo) {
        case fil::leaf_algo_t::FLOAT_UNARY_BINARY:
        case fil::leaf_algo_t::GROVE_PER_CLASS:
          // default is fil::FLOAT_UNARY_BINARY
          builder->SetLeafNode(
            key, tlf::Value::Create(dense_node.base_node::output<val_t>().f));
          break;
        case fil::leaf_algo_t::CATEGORICAL_LEAF:
          std::vector<tlf::Value> vec(ps.num_classes);
          for (int i = 0; i < ps.num_classes; ++i) {
            vec[i] = tlf::Value::Create(
              i == dense_node.base_node::output<val_t>().idx ? 1.0f : 0.0f);
          }
          builder->SetLeafVectorNode(key, vec);
      }
    } else {
      int left = root + 2 * (node - root) + 1;
      int right = root + 2 * (node - root) + 2;
      float threshold = dense_node.thresh();
      bool default_left = dense_node.def_left();
      switch (ps.op) {
        case tl::Operator::kLT:
          break;
        case tl::Operator::kLE:
          // adjust the threshold
          threshold =
            std::nextafterf(threshold, -std::numeric_limits<float>::infinity());
          break;
        case tl::Operator::kGT:
          // adjust the threshold; left and right still need to be swapped
          threshold =
            std::nextafterf(threshold, -std::numeric_limits<float>::infinity());
        case tl::Operator::kGE:
          // swap left and right
          std::swap(left, right);
          default_left = !default_left;
          break;
        default:
          ASSERT(false, "comparison operator must be <, >, <= or >=");
      }
      int left_key = node_to_treelite(builder, pkey, root, left);
      int right_key = node_to_treelite(builder, pkey, root, right);
      builder->SetNumericalTestNode(key, dense_node.fid(), ps.op,
                                    tlf::Value::Create(threshold), default_left,
                                    left_key, right_key);
    }
    return key;
  }

  void init_forest_impl(fil::forest_t* pforest,
                        fil::storage_type_t storage_type) {
    bool random_forest_flag = (ps.output & fil::output_t::AVG) != 0;
    int treelite_num_classes =
      ps.leaf_algo == fil::leaf_algo_t::FLOAT_UNARY_BINARY ? 1 : ps.num_classes;
    std::unique_ptr<tlf::ModelBuilder> model_builder(new tlf::ModelBuilder(
      ps.num_cols, treelite_num_classes, random_forest_flag,
      tl::TypeInfo::kFloat32, tl::TypeInfo::kFloat32));

    // prediction transform
    if ((ps.output & fil::output_t::SIGMOID) != 0) {
      if (ps.num_classes > 2)
        model_builder->SetModelParam("pred_transform", "multiclass_ova");
      else
        model_builder->SetModelParam("pred_transform", "sigmoid");
    } else if (ps.leaf_algo != fil::leaf_algo_t::FLOAT_UNARY_BINARY) {
      model_builder->SetModelParam("pred_transform", "max_index");
      ps.output = fil::output_t(ps.output | fil::output_t::CLASS);
    } else if (ps.leaf_algo == GROVE_PER_CLASS) {
      model_builder->SetModelParam("pred_transform", "identity_multiclass");
    } else {
      model_builder->SetModelParam("pred_transform", "identity");
    }

    // global bias
    char* global_bias_str = nullptr;
    ASSERT(asprintf(&global_bias_str, "%f", double(ps.global_bias)) > 0,
           "cannot convert global_bias into a string");
    model_builder->SetModelParam("global_bias", global_bias_str);
    ::free(global_bias_str);

    // build the trees
    for (int i_tree = 0; i_tree < ps.num_trees; ++i_tree) {
      tlf::TreeBuilder* tree_builder =
        new tlf::TreeBuilder(tl::TypeInfo::kFloat32, tl::TypeInfo::kFloat32);
      int key_counter = 0;
      int root = i_tree * tree_num_nodes();
      int root_key = node_to_treelite(tree_builder, &key_counter, root, root);
      tree_builder->SetRootNode(root_key);
      // InsertTree() consumes tree_builder
      TL_CPP_CHECK(model_builder->InsertTree(tree_builder));
    }

    // commit the model
    std::unique_ptr<tl::Model> model = model_builder->CommitModel();

    // init FIL forest with the model
    fil::treelite_params_t params;
    params.algo = ps.algo;
    params.threshold = ps.threshold;
    params.output_class = (ps.output & fil::output_t::CLASS) != 0;
    params.storage_type = storage_type;
    params.blocks_per_sm = ps.blocks_per_sm;
    fil::from_treelite(handle, pforest, (ModelHandle)model.get(), &params);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
};

class TreeliteDenseFilTest : public TreeliteFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    init_forest_impl(pforest, fil::storage_type_t::DENSE);
  }
};

class TreeliteSparse16FilTest : public TreeliteFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    init_forest_impl(pforest, fil::storage_type_t::SPARSE);
  }
};

class TreeliteSparse8FilTest : public TreeliteFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    init_forest_impl(pforest, fil::storage_type_t::SPARSE8);
  }
};

class TreeliteAutoFilTest : public TreeliteFilTest {
 protected:
  void init_forest(fil::forest_t* pforest) override {
    init_forest_impl(pforest, fil::storage_type_t::AUTO);
  }
};

// test for failures; currently only supported for sparse8 nodes
class TreeliteThrowSparse8FilTest : public TreeliteSparse8FilTest {
 protected:
  // model import happens in check(), so this function is empty
  void SetUp() override {}

  void check() { ASSERT_THROW(setup_helper(), raft::exception); }
};

/** mechanism to use named aggregate initialization before C++20, and also use
    the struct defaults. Using it directly only works if all defaulted
    members come after ones explicitly mentioned.
**/
#define FIL_TEST_PARAMS(...)                                \
  []() {                                                    \
    struct NonDefaultFilTestParams : public FilTestParams { \
      NonDefaultFilTestParams() { __VA_ARGS__; }            \
    };                                                      \
    return FilTestParams(NonDefaultFilTestParams());        \
  }()

// kEQ is intentionally unused, and kLT is default
static const tl::Operator kLE = tl::Operator::kLE;
static const tl::Operator kGT = tl::Operator::kGT;
static const tl::Operator kGE = tl::Operator::kGE;

std::vector<FilTestParams> predict_dense_inputs = {
  FIL_TEST_PARAMS(),
  FIL_TEST_PARAMS(algo = TREE_REORG),
  FIL_TEST_PARAMS(algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(output = SIGMOID),
  FIL_TEST_PARAMS(output = SIGMOID, algo = TREE_REORG),
  FIL_TEST_PARAMS(output = SIGMOID, algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, num_classes = 2),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, algo = TREE_REORG, num_classes = 2),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, algo = BATCH_TREE_REORG,
                  num_classes = 2),
  FIL_TEST_PARAMS(output = AVG),
  FIL_TEST_PARAMS(output = AVG, algo = TREE_REORG),
  FIL_TEST_PARAMS(output = AVG, algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(output = AVG_CLASS, num_classes = 2),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = TREE_REORG, num_classes = 2),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = BATCH_TREE_REORG, num_classes = 2),
  FIL_TEST_PARAMS(global_bias = 0.5, algo = TREE_REORG),
  FIL_TEST_PARAMS(output = SIGMOID, global_bias = 0.5, algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(output = AVG, global_bias = 0.5),
  FIL_TEST_PARAMS(output = AVG_CLASS, threshold = 1.0, global_bias = 0.5,
                  algo = TREE_REORG, num_classes = 2),
  FIL_TEST_PARAMS(output = SIGMOID, algo = ALGO_AUTO),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = BATCH_TREE_REORG,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 5),
  FIL_TEST_PARAMS(output = AVG_CLASS, num_classes = 2),
  FIL_TEST_PARAMS(algo = TREE_REORG, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 5),
  FIL_TEST_PARAMS(output = SIGMOID, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 7),
  FIL_TEST_PARAMS(global_bias = 0.5, algo = TREE_REORG,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 4),
  FIL_TEST_PARAMS(output = AVG, global_bias = 0.5, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 4),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = BATCH_TREE_REORG,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 5),
  FIL_TEST_PARAMS(algo = TREE_REORG, leaf_algo = GROVE_PER_CLASS,
                  num_classes = 5),
  FIL_TEST_PARAMS(num_trees = 49, output = SIGMOID, leaf_algo = GROVE_PER_CLASS,
                  num_classes = 7),
  FIL_TEST_PARAMS(num_trees = 52, global_bias = 0.5, algo = TREE_REORG,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 4),
  FIL_TEST_PARAMS(num_trees = 52, output = AVG, global_bias = 0.5,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 4),
  FIL_TEST_PARAMS(blocks_per_sm = 1),
  FIL_TEST_PARAMS(blocks_per_sm = 4),
  FIL_TEST_PARAMS(num_classes = 3, blocks_per_sm = 1,
                  leaf_algo = CATEGORICAL_LEAF),
  FIL_TEST_PARAMS(num_classes = 3, blocks_per_sm = 4,
                  leaf_algo = CATEGORICAL_LEAF),
  FIL_TEST_PARAMS(num_classes = 5, blocks_per_sm = 1,
                  leaf_algo = GROVE_PER_CLASS),
  FIL_TEST_PARAMS(num_classes = 5, blocks_per_sm = 4,
                  leaf_algo = GROVE_PER_CLASS),
  FIL_TEST_PARAMS(leaf_algo = GROVE_PER_CLASS, blocks_per_sm = 1,
                  num_trees = 512, num_classes = 512),
  FIL_TEST_PARAMS(leaf_algo = GROVE_PER_CLASS, blocks_per_sm = 4,
                  num_trees = 512, num_classes = 512),
  FIL_TEST_PARAMS(num_trees = 52, output = SOFTMAX, leaf_algo = GROVE_PER_CLASS,
                  num_classes = 4),
  FIL_TEST_PARAMS(num_trees = 52, output = AVG_SOFTMAX,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 4),
  FIL_TEST_PARAMS(num_trees = 3 * (FIL_TPB + 1), output = SOFTMAX,
                  leaf_algo = GROVE_PER_CLASS, num_classes = FIL_TPB + 1),
  FIL_TEST_PARAMS(num_trees = 3 * (FIL_TPB + 1), output = AVG_SOFTMAX,
                  leaf_algo = GROVE_PER_CLASS, num_classes = FIL_TPB + 1),
  FIL_TEST_PARAMS(num_cols = 100'000, depth = 5, num_trees = 1,
                  leaf_algo = FLOAT_UNARY_BINARY),
  FIL_TEST_PARAMS(num_rows = 101, num_cols = 100'000, depth = 5, num_trees = 9,
                  algo = BATCH_TREE_REORG, leaf_algo = GROVE_PER_CLASS,
                  num_classes = 3),
  FIL_TEST_PARAMS(num_rows = 102, num_cols = 100'000, depth = 5,
                  num_trees = 3 * (FIL_TPB + 1), algo = BATCH_TREE_REORG,
                  leaf_algo = GROVE_PER_CLASS, num_classes = FIL_TPB + 1),
  FIL_TEST_PARAMS(num_rows = 103, num_cols = 100'000, depth = 5, num_trees = 1,
                  algo = BATCH_TREE_REORG, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 3),
};

TEST_P(PredictDenseFilTest, Predict) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, PredictDenseFilTest,
                        testing::ValuesIn(predict_dense_inputs));

std::vector<FilTestParams> predict_sparse_inputs = {
  FIL_TEST_PARAMS(),
  FIL_TEST_PARAMS(output = SIGMOID),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, num_classes = 2),
  FIL_TEST_PARAMS(output = AVG),
  FIL_TEST_PARAMS(output = AVG_CLASS, global_bias = 0.5, num_classes = 2),
  FIL_TEST_PARAMS(global_bias = 0.5),
  FIL_TEST_PARAMS(output = SIGMOID, global_bias = 0.5),
  FIL_TEST_PARAMS(output = AVG, global_bias = 0.5),
  FIL_TEST_PARAMS(output = AVG_CLASS, threshold = 1.0, global_bias = 0.5,
                  num_classes = 2),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, algo = ALGO_AUTO, num_classes = 2),
  FIL_TEST_PARAMS(output = AVG_CLASS, threshold = 1.0, global_bias = 0.5,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 5000),
  FIL_TEST_PARAMS(global_bias = 0.5, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 6),
  FIL_TEST_PARAMS(output = CLASS, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 3),
  FIL_TEST_PARAMS(leaf_algo = CATEGORICAL_LEAF, num_classes = 3),
  FIL_TEST_PARAMS(depth = 2, num_trees = 5000, output = AVG_CLASS,
                  threshold = 1.0, global_bias = 0.5,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 5000),
  FIL_TEST_PARAMS(num_trees = 60, global_bias = 0.5,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 6),
  FIL_TEST_PARAMS(num_trees = 51, output = CLASS, leaf_algo = GROVE_PER_CLASS,
                  num_classes = 3),
  FIL_TEST_PARAMS(num_trees = 51, leaf_algo = GROVE_PER_CLASS, num_classes = 3),
};

TEST_P(PredictSparse16FilTest, Predict) { compare(); }

// Temporarily disabled, see https://github.com/rapidsai/cuml/issues/3205
INSTANTIATE_TEST_CASE_P(FilTests, PredictSparse16FilTest,
                        testing::ValuesIn(predict_sparse_inputs));

TEST_P(PredictSparse8FilTest, Predict) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, PredictSparse8FilTest,
                        testing::ValuesIn(predict_sparse_inputs));

std::vector<FilTestParams> import_dense_inputs = {
  FIL_TEST_PARAMS(),
  FIL_TEST_PARAMS(output = SIGMOID, op = kLE),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, op = kGT, num_classes = 2),
  FIL_TEST_PARAMS(output = AVG, op = kGE),
  FIL_TEST_PARAMS(output = AVG_CLASS, num_classes = 2),
  FIL_TEST_PARAMS(algo = TREE_REORG, op = kLE),
  FIL_TEST_PARAMS(output = SIGMOID, algo = TREE_REORG, op = kGT),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, algo = TREE_REORG, op = kGE,
                  num_classes = 2),
  FIL_TEST_PARAMS(output = AVG, algo = TREE_REORG),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = TREE_REORG, op = kLE,
                  num_classes = 2),
  FIL_TEST_PARAMS(algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(output = SIGMOID, algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(algo = BATCH_TREE_REORG, op = kLE),
  FIL_TEST_PARAMS(output = SIGMOID, algo = BATCH_TREE_REORG, op = kLE),
  FIL_TEST_PARAMS(algo = BATCH_TREE_REORG, op = kGT),
  FIL_TEST_PARAMS(output = SIGMOID, algo = BATCH_TREE_REORG, op = kGT),
  FIL_TEST_PARAMS(algo = BATCH_TREE_REORG, op = kGE),
  FIL_TEST_PARAMS(output = SIGMOID, algo = BATCH_TREE_REORG, op = kGE),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, algo = BATCH_TREE_REORG,
                  num_classes = 2),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, algo = BATCH_TREE_REORG, op = kLE,
                  num_classes = 2),
  FIL_TEST_PARAMS(output = AVG, algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(output = AVG, algo = BATCH_TREE_REORG, op = kLE),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = BATCH_TREE_REORG, op = kGT,
                  num_classes = 2),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = BATCH_TREE_REORG, op = kGE,
                  num_classes = 2),
  FIL_TEST_PARAMS(global_bias = 0.5, algo = TREE_REORG),
  FIL_TEST_PARAMS(output = SIGMOID, global_bias = 0.5, algo = BATCH_TREE_REORG,
                  op = kLE),
  FIL_TEST_PARAMS(output = AVG, global_bias = 0.5, op = kGT),
  FIL_TEST_PARAMS(output = AVG_CLASS, threshold = 1.0, global_bias = 0.5,
                  algo = TREE_REORG, op = kGE, num_classes = 2),
  FIL_TEST_PARAMS(output = SIGMOID, algo = ALGO_AUTO, op = kLE),
  FIL_TEST_PARAMS(output = SIGMOID, algo = ALGO_AUTO, op = kLE),
  FIL_TEST_PARAMS(output = AVG, algo = BATCH_TREE_REORG, op = kGE,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 5),
  FIL_TEST_PARAMS(output = AVG, algo = BATCH_TREE_REORG, op = kGT,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 6),
  FIL_TEST_PARAMS(output = AVG, algo = BATCH_TREE_REORG, op = kLE,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 3),
  FIL_TEST_PARAMS(output = AVG, algo = BATCH_TREE_REORG, op = kLE,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 5),
  FIL_TEST_PARAMS(output = AVG_CLASS, algo = TREE_REORG, op = kLE,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 5),
  FIL_TEST_PARAMS(output = AVG, algo = TREE_REORG, op = kLE,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 7),
  FIL_TEST_PARAMS(output = AVG, leaf_algo = CATEGORICAL_LEAF, num_classes = 6),
  FIL_TEST_PARAMS(output = CLASS, algo = BATCH_TREE_REORG, op = kGE,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 5),
  FIL_TEST_PARAMS(num_trees = 48, output = CLASS, algo = BATCH_TREE_REORG,
                  op = kGT, leaf_algo = GROVE_PER_CLASS, num_classes = 6),
  FIL_TEST_PARAMS(num_trees = 51, output = CLASS, algo = BATCH_TREE_REORG,
                  op = kLE, leaf_algo = GROVE_PER_CLASS, num_classes = 3),
  FIL_TEST_PARAMS(output = CLASS, algo = BATCH_TREE_REORG, op = kLE,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 5),
  FIL_TEST_PARAMS(output = CLASS, algo = TREE_REORG, op = kLE,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 5),
  FIL_TEST_PARAMS(num_trees = 49, output = CLASS, algo = TREE_REORG, op = kLE,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 7),
  FIL_TEST_PARAMS(num_trees = 48, output = CLASS, leaf_algo = GROVE_PER_CLASS,
                  num_classes = 6),
};

TEST_P(TreeliteDenseFilTest, Import) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteDenseFilTest,
                        testing::ValuesIn(import_dense_inputs));

std::vector<FilTestParams> import_sparse_inputs = {
  FIL_TEST_PARAMS(),
  FIL_TEST_PARAMS(output = SIGMOID, op = kLE),
  FIL_TEST_PARAMS(output = SIGMOID_CLASS, op = kGT, num_classes = 2),
  FIL_TEST_PARAMS(output = AVG, op = kGE),
  FIL_TEST_PARAMS(output = AVG_CLASS, num_classes = 2),
  FIL_TEST_PARAMS(global_bias = 0.5),
  FIL_TEST_PARAMS(output = SIGMOID, global_bias = 0.5, op = kLE),
  FIL_TEST_PARAMS(output = AVG, global_bias = 0.5, op = kGT),
  FIL_TEST_PARAMS(output = AVG_CLASS, threshold = 1.0, global_bias = 0.5,
                  op = kGE, num_classes = 2),
  FIL_TEST_PARAMS(algo = ALGO_AUTO),
  FIL_TEST_PARAMS(output = AVG_CLASS, threshold = 1.0, global_bias = 0.5,
                  op = kGE, leaf_algo = CATEGORICAL_LEAF, num_classes = 10),
  FIL_TEST_PARAMS(output = AVG, algo = ALGO_AUTO, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 4),
  FIL_TEST_PARAMS(output = AVG, op = kLE, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 5),
  FIL_TEST_PARAMS(output = AVG, global_bias = 0.5, leaf_algo = CATEGORICAL_LEAF,
                  num_classes = 3),
  FIL_TEST_PARAMS(output = CLASS, threshold = 1.0, global_bias = 0.5, op = kGE,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 10),
  FIL_TEST_PARAMS(num_trees = 52, output = CLASS, algo = ALGO_AUTO,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 4),
  FIL_TEST_PARAMS(output = CLASS, op = kLE, leaf_algo = GROVE_PER_CLASS,
                  num_classes = 5),
  FIL_TEST_PARAMS(num_trees = 51, output = CLASS, global_bias = 0.5,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 3),
  FIL_TEST_PARAMS(num_trees = 51, output = SIGMOID_CLASS, global_bias = 0.5,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 3),
};

TEST_P(TreeliteSparse16FilTest, Import) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteSparse16FilTest,
                        testing::ValuesIn(import_sparse_inputs));

TEST_P(TreeliteSparse8FilTest, Import) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteSparse8FilTest,
                        testing::ValuesIn(import_sparse_inputs));

std::vector<FilTestParams> import_auto_inputs = {
  FIL_TEST_PARAMS(depth = 10, algo = ALGO_AUTO),
  FIL_TEST_PARAMS(depth = 15, algo = ALGO_AUTO),
  FIL_TEST_PARAMS(depth = 19, algo = ALGO_AUTO),
  FIL_TEST_PARAMS(depth = 19, algo = BATCH_TREE_REORG),
  FIL_TEST_PARAMS(depth = 10, output = AVG, algo = ALGO_AUTO,
                  leaf_algo = CATEGORICAL_LEAF, num_classes = 3),
  FIL_TEST_PARAMS(depth = 10, num_trees = 51, output = CLASS, algo = ALGO_AUTO,
                  leaf_algo = GROVE_PER_CLASS, num_classes = 3),
#if 0
 FIL_TEST_PARAMS(depth = 19, output = AVG, algo = BATCH_TREE_REORG,
                 leaf_algo = CATEGORICAL_LEAF, num_classes = 6),
#endif
};

TEST_P(TreeliteAutoFilTest, Import) { compare(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteAutoFilTest,
                        testing::ValuesIn(import_auto_inputs));

// adjust test parameters if the sparse8 format changes
std::vector<FilTestParams> import_throw_sparse8_inputs = {
  // too many features
  FIL_TEST_PARAMS(num_rows = 100, num_cols = 20000, depth = 10),
  // too many tree nodes
  FIL_TEST_PARAMS(depth = 16, num_trees = 5, leaf_prob = 0),
};

TEST_P(TreeliteThrowSparse8FilTest, Import) { check(); }

INSTANTIATE_TEST_CASE_P(FilTests, TreeliteThrowSparse8FilTest,
                        testing::ValuesIn(import_throw_sparse8_inputs));
}  // namespace ML
