#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/handle.hpp>

#include <decisiontree/quantile/quantile.h>
#include <gtest/gtest.h>
#include <algorithm>
#include <common/iota.cuh>
#include <decisiontree/batched-levelalgo/builder_base.cuh>
#include <decisiontree/batched-levelalgo/kernels.cuh>
#include <decisiontree/batched-levelalgo/metrics.cuh>
#include <functional>

namespace ML {
namespace DecisionTree {

struct NodeSplitKernelTestParams {
  int min_samples_split;
  int min_samples_leaf;
  int expected_n_total_nodes;
  int expected_n_new_nodes;
};

struct NoOpParams {};

class BatchedLevelAlgoUnitTestFixture {
 protected:
  using DataT = float;
  using LabelT = float;
  using IdxT = int;
  using NodeT = Node<DataT, LabelT, IdxT>;
  using SplitT = Split<DataT, IdxT>;
  using InputT = Input<DataT, LabelT, IdxT>;
  using ObjectiveT = MSEObjectiveFunction<DataT, LabelT, IdxT>;

  const int n_bins = 5;
  const IdxT n_row = 5;
  const IdxT n_col = 2;
  const IdxT max_batch = 8;
  static constexpr int TPB_DEFAULT = 256;
  static constexpr int TPB_SPLIT = 128;

  void SetUp() {
    params.max_depth = 2;
    params.max_leaves = 8;
    params.max_features = 1.0f;
    params.n_bins = n_bins;
    params.min_samples_leaf = 0;
    params.min_samples_split = 0;
    params.split_criterion = CRITERION::MSE;
    params.min_impurity_decrease = 0.0f;
    params.max_batch_size = 8;

    h_data = {-1.0f, 0.0f, 2.0f, 0.0f, -2.0f,
              0.0f,  1.0f, 0.0f, 3.0f, 0.0f};  // column-major
    h_labels = {-1.0f, 2.0f, 2.0f, 6.0f, -2.0f};
    // X0 + 2 * X1

    raft_handle = std::make_unique<raft::handle_t>();
    auto d_allocator = raft_handle->get_device_allocator();

    data = static_cast<DataT*>(
      d_allocator->allocate(sizeof(DataT) * n_row * n_col, 0));
    d_quantiles = static_cast<DataT*>(
      d_allocator->allocate(sizeof(DataT) * n_bins * n_col, 0));
    labels =
      static_cast<LabelT*>(d_allocator->allocate(sizeof(LabelT) * n_row, 0));
    row_ids =
      static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT) * n_row, 0));

    // Nodes that exist prior to the invocation of nodeSplitKernel()
    curr_nodes =
      static_cast<NodeT*>(d_allocator->allocate(sizeof(NodeT) * max_batch, 0));
    // Nodes that are created new by the invocation of nodeSplitKernel()
    new_nodes = static_cast<NodeT*>(
      d_allocator->allocate(sizeof(NodeT) * 2 * max_batch, 0));
    // Number of nodes and leaves that are created new by the invocation of
    // nodeSplitKernel()
    n_new_nodes = static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT), 0));
    n_new_leaves = static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT), 0));
    // New depth reached by the invocation of nodeSplitKernel()
    new_depth = static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT), 0));

    splits = static_cast<SplitT*>(
      d_allocator->allocate(sizeof(SplitT) * max_batch, 0));

    raft::update_device(data, h_data.data(), n_row * n_col, 0);
    raft::update_device(labels, h_labels.data(), n_row, 0);
    computeQuantiles(d_quantiles, n_bins, data, n_row, n_col, d_allocator,
                     nullptr);
    MLCommon::iota(row_ids, 0, 1, n_row, 0);

    CUDA_CHECK(hipStreamSynchronize(0));

    input.data = data;
    input.labels = labels;
    input.M = n_row;
    input.N = n_col;
    input.nSampledRows = n_row;
    input.nSampledCols = n_col;
    input.rowids = row_ids;
    input.numOutputs = 1;
    input.quantiles = d_quantiles;
  }

  void TearDown() {
    auto d_allocator = raft_handle->get_device_allocator();
    d_allocator->deallocate(data, sizeof(DataT) * n_row * n_col, 0);
    d_allocator->deallocate(d_quantiles, sizeof(DataT) * n_bins * n_col, 0);
    d_allocator->deallocate(labels, sizeof(LabelT) * n_row, 0);
    d_allocator->deallocate(row_ids, sizeof(IdxT) * n_row, 0);
    d_allocator->deallocate(curr_nodes, sizeof(NodeT) * max_batch, 0);
    d_allocator->deallocate(new_nodes, sizeof(NodeT) * 2 * max_batch, 0);
    d_allocator->deallocate(n_new_nodes, sizeof(IdxT), 0);
    d_allocator->deallocate(n_new_leaves, sizeof(IdxT), 0);
    d_allocator->deallocate(new_depth, sizeof(IdxT), 0);
    d_allocator->deallocate(splits, sizeof(SplitT) * max_batch, 0);
  }

  DecisionTreeParams params;

  std::unique_ptr<raft::handle_t> raft_handle;

  std::vector<DataT> h_data;
  std::vector<LabelT> h_labels;

  DataT* d_quantiles;
  InputT input;

  NodeT* curr_nodes;
  NodeT* new_nodes;
  IdxT* n_new_nodes;
  IdxT* n_new_leaves;
  IdxT* new_depth;
  SplitT* splits;

  DataT* data;
  DataT* labels;
  IdxT* row_ids;
};

class TestNodeSplitKernel
  : public ::testing::TestWithParam<NodeSplitKernelTestParams>,
    protected BatchedLevelAlgoUnitTestFixture {
 protected:
  void SetUp() override { BatchedLevelAlgoUnitTestFixture::SetUp(); }

  void TearDown() override { BatchedLevelAlgoUnitTestFixture::TearDown(); }
};

class TestMetric : public ::testing::TestWithParam<CRITERION>,
                   protected BatchedLevelAlgoUnitTestFixture {
 protected:
  void SetUp() override { BatchedLevelAlgoUnitTestFixture::SetUp(); }

  void TearDown() override { BatchedLevelAlgoUnitTestFixture::TearDown(); }
};

TEST_P(TestNodeSplitKernel, MinSamplesSplitLeaf) {
  auto test_params = GetParam();

  Builder<ObjectiveT> builder;
  builder.input = input;
  auto smemSize = builder.nodeSplitSmemSize();

  IdxT h_n_total_nodes = 3;  // total number of nodes created so far
  IdxT h_n_new_nodes;        // number of nodes created in this round
  IdxT batchSize = 2;
  std::vector<NodeT> h_nodes{
    /* {
     *   SparseTreeNode{
     *     prediction, colid, quesval, best_metric_val, left_child_id },
     *   }, start, count, depth
     * } */
    {{1.40f, 0, -0.5f, 5.606667f, 1}, 0, 5, 0},
    {{-1.50f, IdxT(-1), DataT(0), DataT(0), NodeT::Leaf}, 0, 2, 1},
    {{3.333333f, IdxT(-1), DataT(0), DataT(0), NodeT::Leaf}, 1, 3, 1},
  };
  raft::update_device(curr_nodes, h_nodes.data() + 1, batchSize, 0);
  CUDA_CHECK(hipMemsetAsync(n_new_nodes, 0, sizeof(IdxT), 0));
  CUDA_CHECK(hipMemsetAsync(n_new_leaves, 0, sizeof(IdxT), 0));
  CUDA_CHECK(hipMemsetAsync(new_depth, 0, sizeof(IdxT), 0));
  initSplit<DataT, IdxT, builder.TPB_DEFAULT>(splits, batchSize, 0);

  /* { quesval, colid, best_metric_val, nLeft } */
  std::vector<SplitT> h_splits{{-1.5f, 0, 0.25f, 1}, {2.0f, 1, 3.555556f, 2}};
  raft::update_device(splits, h_splits.data(), 2, 0);

  nodeSplitKernel<DataT, LabelT, IdxT, ObjectiveT, builder.TPB_SPLIT>
    <<<batchSize, builder.TPB_SPLIT, smemSize, 0>>>(
      params.max_depth, test_params.min_samples_leaf,
      test_params.min_samples_split, params.max_leaves,
      params.min_impurity_decrease, input, curr_nodes, new_nodes, n_new_nodes,
      splits, n_new_leaves, h_n_total_nodes, new_depth);
  CUDA_CHECK(hipGetLastError());
  raft::update_host(&h_n_new_nodes, n_new_nodes, 1, 0);
  CUDA_CHECK(hipStreamSynchronize(0));
  h_n_total_nodes += h_n_new_nodes;
  EXPECT_EQ(h_n_total_nodes, test_params.expected_n_total_nodes);
  EXPECT_EQ(h_n_new_nodes, test_params.expected_n_new_nodes);
}

const std::vector<NodeSplitKernelTestParams> min_samples_split_leaf_test_params{
  /* { min_samples_split, min_samples_leaf,
   *   expected_n_total_nodes, expected_n_new_nodes } */
  {0, 0, 7, 4}, {2, 0, 7, 4}, {3, 0, 5, 2}, {4, 0, 3, 0}, {5, 0, 3, 0},
  {0, 1, 7, 4}, {0, 2, 3, 0}, {0, 5, 3, 0}, {4, 2, 3, 0}, {5, 5, 3, 0}};

INSTANTIATE_TEST_SUITE_P(
  BatchedLevelAlgoUnitTest, TestNodeSplitKernel,
  ::testing::ValuesIn(min_samples_split_leaf_test_params));

TEST_P(TestMetric, RegressionMetricGain) {
  IdxT batchSize = 1;
  std::vector<NodeT> h_nodes{
    /* {
     *   SparseTreeNode{
     *     prediction, colid, quesval, best_metric_val, left_child_id },
     *   }, start, count, depth
     * } */
    {{1.40f, IdxT(-1), DataT(0), DataT(0), NodeT::Leaf}, 0, 5, 0}};
  raft::update_device(curr_nodes, h_nodes.data(), batchSize, 0);

  auto n_col_blks = 1;  // evaluate only one column (feature)

  IdxT nPredCounts = max_batch * n_bins * n_col_blks;

  auto d_allocator = raft_handle->get_device_allocator();

  // mutex array used for atomically updating best split
  int* mutex =
    static_cast<int*>(d_allocator->allocate(sizeof(int) * max_batch, 0));
  // threadblock arrival count
  int* done_count = static_cast<int*>(
    d_allocator->allocate(sizeof(int) * max_batch * n_col_blks, 0));
  ObjectiveT::BinT* hist = static_cast<ObjectiveT::BinT*>(
    d_allocator->allocate(2 * nPredCounts * sizeof(ObjectiveT::BinT), 0));

  WorkloadInfo<IdxT>* workload_info = static_cast<WorkloadInfo<IdxT>*>(
    d_allocator->allocate(sizeof(WorkloadInfo<IdxT>), 0));
  WorkloadInfo<IdxT> h_workload_info;

  // Just one threadBlock would be used
  h_workload_info.nodeid = 0;
  h_workload_info.offset_blockid = 0;
  h_workload_info.num_blocks = 1;

  raft::update_device(workload_info, &h_workload_info, 1, 0);
  CUDA_CHECK(hipMemsetAsync(mutex, 0, sizeof(int) * max_batch, 0));
  CUDA_CHECK(
    hipMemsetAsync(done_count, 0, sizeof(int) * max_batch * n_col_blks, 0));
  CUDA_CHECK(hipMemsetAsync(hist, 0, 2 * sizeof(DataT) * nPredCounts, 0));
  CUDA_CHECK(hipMemsetAsync(n_new_leaves, 0, sizeof(IdxT), 0));
  initSplit<DataT, IdxT, TPB_DEFAULT>(splits, batchSize, 0);

  std::vector<SplitT> h_splits(1);

  CRITERION split_criterion = GetParam();

  ObjectiveT obj(1, params.min_impurity_decrease, params.min_samples_leaf);
  size_t smemSize1 = n_bins * sizeof(ObjectiveT::BinT) +  // pdf_shist size
                     n_bins * sizeof(ObjectiveT::BinT) +  // cdf_shist size
                     n_bins * sizeof(DataT) +             // sbins size
                     sizeof(int);                         // sDone size
  // Extra room for alignment (see alignPointer in
  // computeSplitClassificationKernel)
  smemSize1 += sizeof(DataT) + 3 * sizeof(int);
  // Calculate the shared memory needed for evalBestSplit
  size_t smemSize2 =
    raft::ceildiv(TPB_DEFAULT, raft::WarpSize) * sizeof(SplitT);
  // Pick the max of two
  size_t smemSize = std::max(smemSize1, smemSize2);

  dim3 grid(1, n_col_blks, 1);
  computeSplitKernel<DataT, LabelT, IdxT, 32><<<grid, 32, smemSize, 0>>>(
    hist, n_bins, params.max_depth, params.min_samples_split, params.max_leaves,
    input, curr_nodes, 0, done_count, mutex, splits, obj, 0, workload_info,
    1234ULL);

  raft::update_host(h_splits.data(), splits, 1, 0);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipStreamSynchronize(0));

  // the split uses feature 0
  // rows 0, 4 go to the left side of the threshold
  // rows 1, 2, 3 go to the right side of the threshold
  EXPECT_EQ(h_splits[0].colid, 0);
  EXPECT_EQ(h_splits[0].nLeft, 2);
  for (int row_id : {0, 4}) {
    EXPECT_LE(h_data[0 * n_row + row_id], h_splits[0].quesval);
  }
  for (int row_id : {1, 2, 3}) {
    EXPECT_GT(h_data[0 * n_row + row_id], h_splits[0].quesval);
  }
  // Verify that the gain (reduction in MSE / MAE) is computed correctly
  std::function<float(const std::vector<DataT>&, const std::vector<IdxT>&)>
    metric;
  if (split_criterion == CRITERION::MSE) {
    metric = [](const std::vector<DataT>& y,
                const std::vector<IdxT>& idx) -> float {
      float y_mean = 0.0f;
      float mse = 0.0f;
      for (IdxT i : idx) {
        y_mean += y[i];
      }
      y_mean /= idx.size();
      for (IdxT i : idx) {
        mse += (y[i] - y_mean) * (y[i] - y_mean);
      }
      return mse / idx.size();
    };
  } else {
    EXPECT_EQ(split_criterion, CRITERION::MAE);
    metric = [](const std::vector<DataT>& y,
                const std::vector<IdxT>& idx) -> float {
      float y_mean = 0.0f;
      float mae = 0.0f;
      for (IdxT i : idx) {
        y_mean += y[i];
      }
      y_mean /= idx.size();
      for (IdxT i : idx) {
        mae += std::fabs(y[i] - y_mean);
      }
      return mae / idx.size();
    };
  }
  float expected_gain = metric(h_labels, {0, 1, 2, 3, 4}) -
                        2.0f / 5.0f * metric(h_labels, {0, 4}) -
                        3.0f / 5.0f * metric(h_labels, {1, 2, 3});

  EXPECT_FLOAT_EQ(h_splits[0].best_metric_val, expected_gain);

  d_allocator->deallocate(mutex, sizeof(int) * max_batch, 0);
  d_allocator->deallocate(done_count, sizeof(int) * max_batch * n_col_blks, 0);
  d_allocator->deallocate(hist, 2 * nPredCounts * sizeof(DataT), 0);
  d_allocator->deallocate(workload_info, sizeof(WorkloadInfo<IdxT>), 0);
}

INSTANTIATE_TEST_SUITE_P(BatchedLevelAlgoUnitTest, TestMetric,
                         ::testing::Values(CRITERION::MSE),
                         [](const auto& info) {
                           switch (info.param) {
                             case CRITERION::MSE:
                               return "MSE";
                             default:
                               return "";
                           }
                         });

}  // namespace DecisionTree
}  // namespace ML
