#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <decisiontree/memory.h>
#include <decisiontree/quantile/quantile.h>
#include <gtest/gtest.h>
#include <algorithm>
#include <common/iota.cuh>
#include <decisiontree/batched-levelalgo/builder_base.cuh>
#include <decisiontree/batched-levelalgo/kernels.cuh>
#include <functional>

namespace ML {
namespace DecisionTree {

struct NodeSplitKernelTestParams {
  int min_samples_split;
  int min_samples_leaf;
  int expected_n_total_nodes;
  int expected_n_new_nodes;
};

struct NoOpParams {};

class BatchedLevelAlgoUnitTestFixture {
 protected:
  using DataT = float;
  using LabelT = float;
  using IdxT = int;
  using NodeT = Node<DataT, LabelT, IdxT>;
  using Traits = RegTraits<DataT, IdxT>;

  const int n_bins = 5;
  const IdxT n_row = 5;
  const IdxT n_col = 2;
  const IdxT max_batch = 8;

  void SetUp() {
    params.max_depth = 2;
    params.max_leaves = 8;
    params.max_features = 1.0f;
    params.n_bins = n_bins;
    params.split_algo = 1;
    params.min_samples_leaf = 0;
    params.min_samples_split = 0;
    params.bootstrap_features = false;
    params.quantile_per_tree = false;
    params.split_criterion = CRITERION::MSE;
    params.min_impurity_decrease = 0.0f;
    params.max_batch_size = 8;
    params.use_experimental_backend = true;

    h_data = {-1.0f, 0.0f, 2.0f, 0.0f, -2.0f,
              0.0f,  1.0f, 0.0f, 3.0f, 0.0f};  // column-major
    h_labels = {-1.0f, 2.0f, 2.0f, 6.0f, -2.0f};
    // X0 + 2 * X1

    raft_handle = std::make_unique<raft::handle_t>();
    auto d_allocator = raft_handle->get_device_allocator();

    data = static_cast<DataT*>(
      d_allocator->allocate(sizeof(DataT) * n_row * n_col, 0));
    labels =
      static_cast<LabelT*>(d_allocator->allocate(sizeof(LabelT) * n_row, 0));
    row_ids =
      static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT) * n_row, 0));

    // Nodes that exist prior to the invocation of nodeSplitKernel()
    curr_nodes =
      static_cast<NodeT*>(d_allocator->allocate(sizeof(NodeT) * max_batch, 0));
    // Nodes that are created new by the invocation of nodeSplitKernel()
    new_nodes = static_cast<NodeT*>(
      d_allocator->allocate(sizeof(NodeT) * 2 * max_batch, 0));
    // Number of nodes and leaves that are created new by the invocation of
    // nodeSplitKernel()
    n_new_nodes = static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT), 0));
    n_new_leaves = static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT), 0));
    // New depth reached by the invocation of nodeSplitKernel()
    new_depth = static_cast<IdxT*>(d_allocator->allocate(sizeof(IdxT), 0));

    splits = static_cast<Traits::SplitT*>(
      d_allocator->allocate(sizeof(Traits::SplitT) * max_batch, 0));

    raft::update_device(data, h_data.data(), n_row * n_col, 0);
    raft::update_device(labels, h_labels.data(), n_row, 0);
    MLCommon::iota(row_ids, 0, 1, n_row, 0);

    tempmem = std::make_shared<TemporaryMemory<DataT, LabelT>>(
      *raft_handle, hipStream_t(0), n_row, n_col, 0, params);
    preprocess_quantile(data, reinterpret_cast<unsigned int*>(row_ids), n_row,
                        n_col, n_row, n_bins, tempmem);
    DataT* quantiles = tempmem->d_quantile->data();
    CUDA_CHECK(hipStreamSynchronize(0));

    h_quantiles = tempmem->h_quantile->data();

    input.data = data;
    input.labels = labels;
    input.M = n_row;
    input.N = n_col;
    input.nSampledRows = n_row;
    input.nSampledCols = n_col;
    input.rowids = row_ids;
    input.nclasses = 0;  // not applicable for regression
    input.quantiles = quantiles;
  }

  void TearDown() {
    auto d_allocator = raft_handle->get_device_allocator();
    d_allocator->deallocate(data, sizeof(DataT) * n_row * n_col, 0);
    d_allocator->deallocate(labels, sizeof(LabelT) * n_row, 0);
    d_allocator->deallocate(row_ids, sizeof(IdxT) * n_row, 0);
    d_allocator->deallocate(curr_nodes, sizeof(NodeT) * max_batch, 0);
    d_allocator->deallocate(new_nodes, sizeof(NodeT) * 2 * max_batch, 0);
    d_allocator->deallocate(n_new_nodes, sizeof(IdxT), 0);
    d_allocator->deallocate(n_new_leaves, sizeof(IdxT), 0);
    d_allocator->deallocate(new_depth, sizeof(IdxT), 0);
    d_allocator->deallocate(splits, sizeof(Traits::SplitT) * max_batch, 0);
  }

  DecisionTreeParams params;

  std::unique_ptr<raft::handle_t> raft_handle;
  std::shared_ptr<TemporaryMemory<DataT, LabelT>> tempmem;

  std::vector<DataT> h_data;
  std::vector<LabelT> h_labels;

  DataT* h_quantiles;
  Traits::InputT input;

  NodeT* curr_nodes;
  NodeT* new_nodes;
  IdxT* n_new_nodes;
  IdxT* n_new_leaves;
  IdxT* new_depth;
  Traits::SplitT* splits;

  DataT* data;
  DataT* labels;
  IdxT* row_ids;
};

class TestQuantiles : public ::testing::TestWithParam<NoOpParams>,
                      protected BatchedLevelAlgoUnitTestFixture {
 protected:
  void SetUp() override { BatchedLevelAlgoUnitTestFixture::SetUp(); }

  void TearDown() override { BatchedLevelAlgoUnitTestFixture::TearDown(); }
};

class TestNodeSplitKernel
  : public ::testing::TestWithParam<NodeSplitKernelTestParams>,
    protected BatchedLevelAlgoUnitTestFixture {
 protected:
  void SetUp() override { BatchedLevelAlgoUnitTestFixture::SetUp(); }

  void TearDown() override { BatchedLevelAlgoUnitTestFixture::TearDown(); }
};

class TestMetric : public ::testing::TestWithParam<CRITERION>,
                   protected BatchedLevelAlgoUnitTestFixture {
 protected:
  void SetUp() override { BatchedLevelAlgoUnitTestFixture::SetUp(); }

  void TearDown() override { BatchedLevelAlgoUnitTestFixture::TearDown(); }
};

TEST_P(TestQuantiles, Quantiles) {
  /* Ensure that quantiles are computed correctly */
  std::vector<DataT> expected_quantiles[]{{-2.0f, -1.0f, 0.0f, 2.0f},
                                          {0.0f, 1.0f, 3.0f}};
  for (int col = 0; col < n_col; col++) {
    std::vector<DataT> col_quantile(n_bins);
    std::copy(h_quantiles + n_bins * col, h_quantiles + n_bins * (col + 1),
              col_quantile.begin());
    auto last = std::unique(col_quantile.begin(), col_quantile.end());
    col_quantile.erase(last, col_quantile.end());
    EXPECT_EQ(col_quantile, expected_quantiles[col]);
  }
}

INSTANTIATE_TEST_SUITE_P(BatchedLevelAlgoUnitTest, TestQuantiles,
                         ::testing::Values(NoOpParams{}));

TEST_P(TestNodeSplitKernel, MinSamplesSplitLeaf) {
  auto test_params = GetParam();

  Builder<Traits> builder;
  auto smemSize = Traits::nodeSplitSmemSize(builder);

  IdxT h_n_total_nodes = 3;  // total number of nodes created so far
  IdxT h_n_new_nodes;        // number of nodes created in this round
  IdxT batchSize = 2;
  std::vector<NodeT> h_nodes{
    /* {
     *   SparseTreeNode{
     *     prediction, colid, quesval, best_metric_val, left_child_id },
     *   }, start, count, depth
     * } */
    {{1.40f, 0, -0.5f, 5.606667f, 1}, 0, 5, 0},
    {{-1.50f, IdxT(-1), DataT(0), DataT(0), NodeT::Leaf}, 0, 2, 1},
    {{3.333333f, IdxT(-1), DataT(0), DataT(0), NodeT::Leaf}, 1, 3, 1},
  };
  raft::update_device(curr_nodes, h_nodes.data() + 1, batchSize, 0);
  CUDA_CHECK(hipMemsetAsync(n_new_nodes, 0, sizeof(IdxT), 0));
  CUDA_CHECK(hipMemsetAsync(n_new_leaves, 0, sizeof(IdxT), 0));
  CUDA_CHECK(hipMemsetAsync(new_depth, 0, sizeof(IdxT), 0));
  initSplit<DataT, IdxT, Traits::TPB_DEFAULT>(splits, batchSize, 0);

  /* { quesval, colid, best_metric_val, nLeft } */
  std::vector<Traits::SplitT> h_splits{{-1.5f, 0, 0.25f, 1},
                                       {2.0f, 1, 3.555556f, 2}};
  raft::update_device(splits, h_splits.data(), 2, 0);

  nodeSplitKernel<DataT, LabelT, IdxT, Traits::DevTraits, Traits::TPB_SPLIT>
    <<<batchSize, Traits::TPB_SPLIT, smemSize, 0>>>(
      params.max_depth, test_params.min_samples_leaf,
      test_params.min_samples_split, params.max_leaves,
      params.min_impurity_decrease, input, curr_nodes, new_nodes, n_new_nodes,
      splits, n_new_leaves, h_n_total_nodes, new_depth);
  CUDA_CHECK(hipGetLastError());
  raft::update_host(&h_n_new_nodes, n_new_nodes, 1, 0);
  CUDA_CHECK(hipStreamSynchronize(0));
  h_n_total_nodes += h_n_new_nodes;
  EXPECT_EQ(h_n_total_nodes, test_params.expected_n_total_nodes);
  EXPECT_EQ(h_n_new_nodes, test_params.expected_n_new_nodes);
}

const std::vector<NodeSplitKernelTestParams> min_samples_split_leaf_test_params{
  /* { min_samples_split, min_samples_leaf,
   *   expected_n_total_nodes, expected_n_new_nodes } */
  {0, 0, 7, 4}, {2, 0, 7, 4}, {3, 0, 5, 2}, {4, 0, 3, 0}, {5, 0, 3, 0},
  {0, 1, 7, 4}, {0, 2, 3, 0}, {0, 5, 3, 0}, {4, 2, 3, 0}, {5, 5, 3, 0}};

INSTANTIATE_TEST_SUITE_P(
  BatchedLevelAlgoUnitTest, TestNodeSplitKernel,
  ::testing::ValuesIn(min_samples_split_leaf_test_params));

TEST_P(TestMetric, RegressionMetricGain) {
  IdxT batchSize = 1;
  std::vector<NodeT> h_nodes{
    /* {
     *   SparseTreeNode{
     *     prediction, colid, quesval, best_metric_val, left_child_id },
     *   }, start, count, depth
     * } */
    {{1.40f, IdxT(-1), DataT(0), DataT(0), NodeT::Leaf}, 0, 5, 0}};
  raft::update_device(curr_nodes, h_nodes.data(), batchSize, 0);

  int n_blks_for_rows = 1;
  auto n_col_blks = 1;  // evaluate only one column (feature)
  dim3 grid(n_blks_for_rows, n_col_blks, batchSize);
  size_t smemSize = 7 * n_bins * sizeof(DataT) + n_bins * sizeof(int);
  smemSize += sizeof(int);
  // Room for alignment in worst case
  smemSize += 5 * sizeof(DataT) + 2 * sizeof(int);

  IdxT nPredCounts = max_batch * n_bins * n_col_blks;
  size_t block_sync_size = MLCommon::GridSync::computeWorkspaceSize(
    dim3(n_blks_for_rows, n_col_blks, max_batch), MLCommon::SyncType::ACROSS_X,
    false);

  auto d_allocator = raft_handle->get_device_allocator();

  // mutex array used for atomically updating best split
  int* mutex =
    static_cast<int*>(d_allocator->allocate(sizeof(int) * max_batch, 0));
  // threadblock arrival count
  int* done_count = static_cast<int*>(
    d_allocator->allocate(sizeof(int) * max_batch * n_col_blks, 0));
  // used for synching across blocks in a kernel
  char* block_sync = static_cast<char*>(
    d_allocator->allocate(sizeof(char) * block_sync_size, 0));
  DataT* pred = static_cast<DataT*>(
    d_allocator->allocate(2 * nPredCounts * sizeof(DataT), 0));
  DataT* pred2 = static_cast<DataT*>(
    d_allocator->allocate(2 * nPredCounts * sizeof(DataT), 0));
  DataT* pred2P =
    static_cast<DataT*>(d_allocator->allocate(nPredCounts * sizeof(DataT), 0));
  IdxT* pred_count =
    static_cast<IdxT*>(d_allocator->allocate(nPredCounts * sizeof(IdxT), 0));
  CUDA_CHECK(hipMemsetAsync(mutex, 0, sizeof(int) * max_batch, 0));
  CUDA_CHECK(
    hipMemsetAsync(done_count, 0, sizeof(int) * max_batch * n_col_blks, 0));
  CUDA_CHECK(hipMemsetAsync(block_sync, 0, sizeof(char) * block_sync_size, 0));
  CUDA_CHECK(hipMemsetAsync(pred, 0, 2 * sizeof(DataT) * nPredCounts, 0));
  CUDA_CHECK(hipMemsetAsync(pred2, 0, sizeof(DataT) * nPredCounts * 2, 0));
  CUDA_CHECK(hipMemsetAsync(pred2P, 0, sizeof(DataT) * nPredCounts, 0));
  CUDA_CHECK(hipMemsetAsync(pred_count, 0, nPredCounts * sizeof(IdxT), 0));
  CUDA_CHECK(hipMemsetAsync(n_new_leaves, 0, sizeof(IdxT), 0));
  initSplit<DataT, IdxT, Traits::TPB_DEFAULT>(splits, batchSize, 0);

  std::vector<Traits::SplitT> h_splits(1);

  CRITERION split_criterion = GetParam();

  computeSplitRegressionKernel<DataT, DataT, IdxT, 32>
    <<<grid, 32, smemSize, 0>>>(
      pred, pred2, pred2P, pred_count, n_bins, params.max_depth,
      params.min_samples_split, params.min_samples_leaf,
      params.min_impurity_decrease, params.max_leaves, input, curr_nodes, 0,
      done_count, mutex, n_new_leaves, splits, block_sync, split_criterion, 0,
      1234ULL);

  raft::update_host(h_splits.data(), splits, 1, 0);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipStreamSynchronize(0));

  // the split uses feature 0
  // rows 0, 4 go to the left side of the threshold
  // rows 1, 2, 3 go to the right side of the threshold
  EXPECT_EQ(h_splits[0].colid, 0);
  EXPECT_EQ(h_splits[0].nLeft, 2);
  for (int row_id : {0, 4}) {
    EXPECT_LE(h_data[0 * n_row + row_id], h_splits[0].quesval);
  }
  for (int row_id : {1, 2, 3}) {
    EXPECT_GT(h_data[0 * n_row + row_id], h_splits[0].quesval);
  }
  // Verify that the gain (reduction in MSE / MAE) is computed correctly
  std::function<float(const std::vector<DataT>&, const std::vector<IdxT>&)>
    metric;
  if (split_criterion == CRITERION::MSE) {
    metric = [](const std::vector<DataT>& y,
                const std::vector<IdxT>& idx) -> float {
      float y_mean = 0.0f;
      float mse = 0.0f;
      for (IdxT i : idx) {
        y_mean += y[i];
      }
      y_mean /= idx.size();
      for (IdxT i : idx) {
        mse += (y[i] - y_mean) * (y[i] - y_mean);
      }
      return mse / idx.size();
    };
  } else {
    EXPECT_EQ(split_criterion, CRITERION::MAE);
    metric = [](const std::vector<DataT>& y,
                const std::vector<IdxT>& idx) -> float {
      float y_mean = 0.0f;
      float mae = 0.0f;
      for (IdxT i : idx) {
        y_mean += y[i];
      }
      y_mean /= idx.size();
      for (IdxT i : idx) {
        mae += std::fabs(y[i] - y_mean);
      }
      return mae / idx.size();
    };
  }
  float expected_gain = metric(h_labels, {0, 1, 2, 3, 4}) -
                        2.0f / 5.0f * metric(h_labels, {0, 4}) -
                        3.0f / 5.0f * metric(h_labels, {1, 2, 3});

  EXPECT_FLOAT_EQ(h_splits[0].best_metric_val, expected_gain);

  d_allocator->deallocate(mutex, sizeof(int) * max_batch, 0);
  d_allocator->deallocate(done_count, sizeof(int) * max_batch * n_col_blks, 0);
  d_allocator->deallocate(block_sync, sizeof(char) * block_sync_size, 0);
  d_allocator->deallocate(pred, 2 * nPredCounts * sizeof(DataT), 0);
  d_allocator->deallocate(pred2, 2 * nPredCounts * sizeof(DataT), 0);
  d_allocator->deallocate(pred2P, nPredCounts * sizeof(DataT), 0);
  d_allocator->deallocate(pred_count, nPredCounts * sizeof(IdxT), 0);
}

INSTANTIATE_TEST_SUITE_P(BatchedLevelAlgoUnitTest, TestMetric,
                         ::testing::Values(CRITERION::MSE, CRITERION::MAE),
                         [](const auto& info) {
                           switch (info.param) {
                             case CRITERION::MSE:
                               return "MSE";
                             case CRITERION::MAE:
                               return "MAE";
                             default:
                               return "";
                           }
                         });

}  // namespace DecisionTree
}  // namespace ML
