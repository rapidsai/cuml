#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <cuml/ensemble/randomforest.hpp>
#include <queue>
#include <raft/cuda_utils.cuh>
#include <random>

namespace ML {

using namespace MLCommon;

template <typename T>  // template useless for now.
struct RfInputs {
  int n_rows;
  int n_cols;
  int n_trees;
  float max_features;
  float max_samples;
  int max_depth;
  int max_leaves;
  bool bootstrap;
  bool bootstrap_features;
  int n_bins;
  int split_algo;
  int min_samples_leaf;
  int min_samples_split;
  float min_impurity_decrease;
  int n_streams;
  CRITERION split_criterion;
};

template <typename T>
class RfClassifierDepthTest : public ::testing::TestWithParam<int> {
 protected:
  void basicTest() {
    const int max_depth = ::testing::TestWithParam<int>::GetParam();
    params = RfInputs<T>{5000,
                         10,
                         1,
                         1.0f,
                         1.0f,
                         max_depth,
                         -1,
                         false,
                         false,
                         8,
                         SPLIT_ALGO::GLOBAL_QUANTILE,
                         2,
                         2,
                         0.0,
                         2,
                         CRITERION::ENTROPY};

    RF_params rf_params;
    rf_params = set_rf_params(
      params.max_depth, params.max_leaves, params.max_features, params.n_bins,
      params.split_algo, params.min_samples_leaf, params.min_samples_split,
      params.min_impurity_decrease, params.bootstrap_features, params.bootstrap,
      params.n_trees, params.max_samples, 0, params.split_criterion, false,
      params.n_streams, false, 128);

    int data_len = params.n_rows * params.n_cols;
    raft::allocate(data, data_len);
    raft::allocate(labels, params.n_rows);

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Populate data (assume Col major)
    std::mt19937 gen(0);
    std::vector<T> data_h(data_len);
    std::normal_distribution<> d{0, 1};
    for (int col = 0; col < params.n_cols; ++col) {
      for (int row = 0; row < params.n_rows; ++row) {
        data_h[row + col * params.n_rows] = d(gen);
      }
    }
    raft::update_device(data, data_h.data(), data_len, stream);

    // Populate labels
    labels_h.resize(params.n_rows);
    for (int row = 0; row < params.n_rows; ++row) {
      labels_h[row] =
        (data_h[row + 2 * params.n_rows] * data_h[row + 3 * params.n_rows] >
         0.5);
    }
    preprocess_labels(params.n_rows, labels_h, labels_map);
    raft::update_device(labels, labels_h.data(), params.n_rows, stream);

    forest = new typename ML::RandomForestMetaData<T, int>;
    null_trees_ptr(forest);

    raft::handle_t handle(rf_params.n_streams);
    handle.set_stream(stream);

    fit(handle, forest, data, params.n_rows, params.n_cols, labels,
        labels_map.size(), rf_params);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    labels_h.clear();
    labels_map.clear();

    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(data));
    delete forest;
  }

 protected:
  RfInputs<T> params;
  T* data;
  int* labels;
  std::vector<int> labels_h;
  std::map<int, int> labels_map;
  // unique map of labels to int vals starting from 0

  RandomForestMetaData<T, int>* forest;
};

template <typename T>
class RfRegressorDepthTest : public ::testing::TestWithParam<int> {
 protected:
  void basicTest() {
    const int max_depth = ::testing::TestWithParam<int>::GetParam();
    params = RfInputs<T>{5000,
                         10,
                         1,
                         1.0f,
                         1.0f,
                         max_depth,
                         -1,
                         false,
                         false,
                         8,
                         SPLIT_ALGO::GLOBAL_QUANTILE,
                         2,
                         2,
                         0.0,
                         2,
                         CRITERION::MSE};

    RF_params rf_params;
    rf_params = set_rf_params(
      params.max_depth, params.max_leaves, params.max_features, params.n_bins,
      params.split_algo, params.min_samples_leaf, params.min_samples_split,
      params.min_impurity_decrease, params.bootstrap_features, params.bootstrap,
      params.n_trees, params.max_samples, 0, params.split_criterion, false,
      params.n_streams, false, 128);

    int data_len = params.n_rows * params.n_cols;
    raft::allocate(data, data_len);
    raft::allocate(labels, params.n_rows);

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Populate data (assume Col major)
    std::mt19937 gen(0);
    std::vector<T> data_h(data_len);
    std::normal_distribution<> d{0, 1};
    for (int col = 0; col < params.n_cols; ++col) {
      for (int row = 0; row < params.n_rows; ++row) {
        data_h[row + col * params.n_rows] = d(gen);
      }
    }
    raft::update_device(data, data_h.data(), data_len, stream);

    // Populate labels
    labels_h.resize(params.n_rows);
    for (int row = 0; row < params.n_rows; ++row) {
      labels_h[row] =
        (data_h[row + 2 * params.n_rows] * data_h[row + 3 * params.n_rows]);
    }
    raft::update_device(labels, labels_h.data(), params.n_rows, stream);

    forest = new typename ML::RandomForestMetaData<T, T>;
    null_trees_ptr(forest);

    raft::handle_t handle(rf_params.n_streams);
    handle.set_stream(stream);

    fit(handle, forest, data, params.n_rows, params.n_cols, labels, rf_params);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    labels_h.clear();

    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(data));
    delete forest;
  }

 protected:
  RfInputs<T> params;
  T* data;
  T* labels;
  std::vector<T> labels_h;

  RandomForestMetaData<T, T>* forest;
};

template <typename L, typename T>
int MaxDepthOfDecisionTree(const DecisionTree::TreeMetaDataNode<T, L>* tree) {
  const auto& node_array = tree->sparsetree;
  std::queue<std::pair<int, int>> q;  // (node ID, depth)
  // Traverse the tree breadth-first
  int initial_depth = 0;
  q.emplace(0, initial_depth);
  int max_depth = initial_depth;
  while (!q.empty()) {
    int node_id, depth;
    std::tie(node_id, depth) = q.front();
    q.pop();
    max_depth = std::max(depth, max_depth);
    const SparseTreeNode<T, L>& node = node_array.at(node_id);
    if (node.colid != -1) {
      q.emplace(node.left_child_id, depth + 1);
      q.emplace(node.left_child_id + 1, depth + 1);
    }
  }
  return max_depth;
}

typedef RfClassifierDepthTest<float> RfClassifierDepthTestF;
TEST_P(RfClassifierDepthTestF, Fit) {
  CUML_LOG_INFO("Param max_depth = %d", params.max_depth);
  for (int i = 0; i < forest->rf_params.n_trees; i++) {
    int actual_max_depth = MaxDepthOfDecisionTree(&(forest->trees[i]));
    ASSERT_EQ(actual_max_depth, params.max_depth);
    ASSERT_EQ(actual_max_depth, forest->trees[i].depth_counter);
  }
}

typedef RfClassifierDepthTest<double> RfClassifierDepthTestD;
TEST_P(RfClassifierDepthTestD, Fit) {
  CUML_LOG_INFO("Param max_depth = %d", params.max_depth);
  for (int i = 0; i < forest->rf_params.n_trees; i++) {
    int actual_max_depth = MaxDepthOfDecisionTree(&(forest->trees[i]));
    ASSERT_EQ(actual_max_depth, params.max_depth);
    ASSERT_EQ(actual_max_depth, forest->trees[i].depth_counter);
  }
}

INSTANTIATE_TEST_CASE_P(RfClassifierDepthTests, RfClassifierDepthTestF,
                        ::testing::Range(0, 19));

INSTANTIATE_TEST_CASE_P(RfClassifierDepthTests, RfClassifierDepthTestD,
                        ::testing::Range(0, 19));

typedef RfRegressorDepthTest<float> RfRegressorDepthTestF;
TEST_P(RfRegressorDepthTestF, Fit) {
  CUML_LOG_INFO("Param max_depth = %d", params.max_depth);
  for (int i = 0; i < forest->rf_params.n_trees; i++) {
    int actual_max_depth = MaxDepthOfDecisionTree(&(forest->trees[i]));
    ASSERT_EQ(actual_max_depth, params.max_depth);
    ASSERT_EQ(actual_max_depth, forest->trees[i].depth_counter);
  }
}

typedef RfRegressorDepthTest<double> RfRegressorDepthTestD;
TEST_P(RfRegressorDepthTestD, Fit) {
  CUML_LOG_INFO("Param max_depth = %d", params.max_depth);
  for (int i = 0; i < forest->rf_params.n_trees; i++) {
    int actual_max_depth = MaxDepthOfDecisionTree(&(forest->trees[i]));
    ASSERT_EQ(actual_max_depth, params.max_depth);
    ASSERT_EQ(actual_max_depth, forest->trees[i].depth_counter);
  }
}

INSTANTIATE_TEST_CASE_P(RfRegressorDepthTests, RfRegressorDepthTestF,
                        ::testing::Range(0, 19));

INSTANTIATE_TEST_CASE_P(RfRegressorDepthTests, RfRegressorDepthTestD,
                        ::testing::Range(0, 19));

}  // end namespace ML
