/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuda_utils.h>
#include <decisiontree/quantile/quantile.h>
#include <gtest/gtest.h>
#include <linalg/cublas_wrappers.h>
#include <random/make_blobs.h>
#include <random/make_regression.h>
#include <test_utils.h>
#include <common/cumlHandle.hpp>
#include <common/iota.cuh>
#include <cuml/cuml.hpp>
#include <decisiontree/batched-levelalgo/builder.cuh>
#include <memory>

namespace ML {
namespace DecisionTree {

struct DtTestParams {
  int M, N, nclasses, max_depth, nbins;
  float min_gain;
  CRITERION splitType;
  unsigned long long seed;
};

::std::ostream& operator<<(::std::ostream& os, const DtTestParams& dims) {
  return os;
}

template <typename T, typename L, typename I = int>
class DtBaseTest : public ::testing::TestWithParam<DtTestParams> {
 protected:
  void SetUp() {
    inparams = ::testing::TestWithParam<DtTestParams>::GetParam();
    handle.reset(new cumlHandle);
    CUDA_CHECK(hipStreamCreate(&stream));
    handle->setStream(stream);
    set_tree_params(params, inparams.max_depth, 1 << inparams.max_depth, 1.f,
                    inparams.nbins, SPLIT_ALGO::GLOBAL_QUANTILE, inparams.nbins,
                    inparams.min_gain, false, inparams.splitType, false, false,
                    128, 10, 4, 0);
    auto allocator = handle->getImpl().getDeviceAllocator();
    data = (T*)allocator->allocate(sizeof(T) * inparams.M * inparams.N, stream);
    labels = (L*)allocator->allocate(sizeof(L) * inparams.M, stream);
    auto* tmp =
      (T*)allocator->allocate(sizeof(T) * inparams.M * inparams.N, stream);
    prepareDataset(tmp);
    auto alpha = T(1.0), beta = T(0.0);
    auto cublas = handle->getImpl().getCublasHandle();
    CUBLAS_CHECK(MLCommon::LinAlg::cublasgeam(
      cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, inparams.M, inparams.N, &alpha, tmp,
      inparams.N, &beta, tmp, inparams.M, data, inparams.M, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    allocator->deallocate(tmp, sizeof(T) * inparams.M * inparams.N, stream);
    rowids = (I*)allocator->allocate(sizeof(I) * inparams.M, stream);
    MLCommon::iota(rowids, 0, 1, inparams.M, stream);
    colids = (I*)allocator->allocate(sizeof(I) * inparams.N, stream);
    MLCommon::iota(colids, 0, 1, inparams.N, stream);
    quantiles =
      (T*)allocator->allocate(sizeof(T) * inparams.nbins * inparams.N, stream);
    preprocess_quantile<T>((const T*)data, (const unsigned*)rowids, inparams.M,
                           inparams.N, inparams.M, inparams.nbins, (T*)nullptr,
                           quantiles, (T*)nullptr, allocator, stream);
  }

  void TearDown() {
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto allocator = handle->getImpl().getDeviceAllocator();
    allocator->deallocate(data, sizeof(T) * inparams.M * inparams.N, stream);
    allocator->deallocate(labels, sizeof(L) * inparams.M, stream);
    allocator->deallocate(rowids, sizeof(int) * inparams.M, stream);
    allocator->deallocate(colids, sizeof(int) * inparams.N, stream);
    allocator->deallocate(quantiles, sizeof(T) * inparams.nbins * inparams.N,
                          stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    handle.reset();
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  hipStream_t stream;
  std::shared_ptr<cumlHandle> handle;
  T *data, *quantiles;
  L* labels;
  I *rowids, *colids;
  DecisionTreeParams params;
  DtTestParams inparams;
  std::vector<SparseTreeNode<T, L>> sparsetree;

  virtual void prepareDataset(T* tmp) = 0;
};  // class DtBaseTest

const std::vector<DtTestParams> allC = {
  {1024, 4, 2, 8, 16, 0.00001f, CRITERION::GINI, 12345ULL},
  {1024, 4, 2, 8, 16, 0.00001f, CRITERION::GINI, 12345ULL},
  {1024, 4, 2, 8, 16, 0.00001f, CRITERION::ENTROPY, 12345ULL},
  {1024, 4, 2, 8, 16, 0.00001f, CRITERION::ENTROPY, 12345ULL},
};
template <typename T>
class DtClassifierTest : public DtBaseTest<T, int> {
 protected:
  void prepareDataset(T* tmp) override {
    auto allocator = this->handle->getImpl().getDeviceAllocator();
    auto inparams = this->inparams;
    MLCommon::Random::make_blobs<T>(tmp, this->labels, inparams.M, inparams.N,
                                    inparams.nclasses, allocator, this->stream,
                                    nullptr, nullptr, T(1.0), false, T(10.0),
                                    T(-10.0), inparams.seed);
  }
};  // class DtClassifierTest
typedef DtClassifierTest<float> DtClsTestF;
///@todo: add checks
TEST_P(DtClsTestF, Test) {
  auto& impl = handle->getImpl();
  int num_leaves, depth;
  grow_tree<float, int, int>(impl.getDeviceAllocator(), impl.getHostAllocator(),
                             data, inparams.N, inparams.M, labels, quantiles,
                             rowids, colids, inparams.M, inparams.nclasses,
                             params, stream, sparsetree, num_leaves, depth);
  // dataset is such that it makes the tree reach all the way to max_depth
  ASSERT_EQ(depth, inparams.max_depth);
}
INSTANTIATE_TEST_CASE_P(BatchedLevelAlgo, DtClsTestF,
                        ::testing::ValuesIn(allC));

const std::vector<DtTestParams> allR = {
  {1024, 4, 2, 8, 16, 0.00001f, CRITERION::MSE, 12345ULL},
  {1024, 4, 2, 8, 16, 0.00001f, CRITERION::MSE, 12345ULL},
};
template <typename T>
class DtRegressorTest : public DtBaseTest<T, T> {
 protected:
  void prepareDataset(T* tmp) override {
    auto allocator = this->handle->getImpl().getDeviceAllocator();
    auto cublas = this->handle->getImpl().getCublasHandle();
    auto cusolver = this->handle->getImpl().getcusolverDnHandle();
    auto inparams = this->inparams;
    MLCommon::Random::make_regression<T>(
      tmp, this->labels, inparams.M, inparams.N, inparams.N, cublas, cusolver,
      allocator, this->stream, nullptr, 1, T(1.0), -1, T(0.5), T(0.0), false,
      inparams.seed);
  }
};  // class DtRegressorTest
typedef DtRegressorTest<float> DtRegTestF;
///@todo: add checks
TEST_P(DtRegTestF, Test) {
  auto& impl = handle->getImpl();
  int num_leaves, depth;
  grow_tree<float, int>(impl.getDeviceAllocator(), impl.getHostAllocator(),
                        data, inparams.N, inparams.M, labels, quantiles, rowids,
                        colids, inparams.M, 0, params, stream, sparsetree,
                        num_leaves, depth);
  ASSERT_EQ(sparsetree.size(), 1);
  ASSERT_EQ(num_leaves, 1);
  ASSERT_EQ(depth, 0);
}
INSTANTIATE_TEST_CASE_P(BatchedLevelAlgo, DtRegTestF,
                        ::testing::ValuesIn(allR));

}  // namespace DecisionTree
}  // end namespace ML
