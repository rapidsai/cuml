#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <test_utils.h>

#include <umap/runner.cuh>

#include <cuml/datasets/make_blobs.hpp>
#include <cuml/manifold/umap.hpp>
#include <cuml/manifold/umapparams.h>
#include <cuml/metrics/metrics.hpp>
#include <cuml/neighbors/knn.hpp>
#include <datasets/digits.h>
#include <raft/cudart_utils.h>
#include <test_utils.h>

#include <datasets/digits.h>
#include <raft/linalg/reduce_rows_by_key.cuh>
#include <selection/knn.cuh>

#include <raft/cuda_utils.cuh>
#include <raft/cudart_utils.h>
#include <raft/distance/distance.hpp>
#include <raft/handle.hpp>
#include <selection/knn.cuh>
#include <umap/runner.cuh>

#include <gtest/gtest.h>

#include <cstddef>
#include <iostream>
#include <type_traits>
#include <vector>

using namespace ML;
using namespace ML::Metrics;

using namespace std;

using namespace MLCommon;
using namespace MLCommon::Datasets::Digits;

template <typename T>
__global__ void has_nan_kernel(T* data, size_t len, bool* answer)
{
  static_assert(std::is_floating_point<T>());
  std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if ((tid < len) && isnan(data[tid])) { *answer = true; }
}

template <typename T>
bool has_nan(T* data, size_t len, hipStream_t stream)
{
  dim3 blk(256);
  dim3 grid(raft::ceildiv(len, (size_t)blk.x));
  bool h_answer = false;
  rmm::device_scalar<bool> d_answer(stream);
  raft::update_device(d_answer.data(), &h_answer, 1, stream);
  has_nan_kernel<<<grid, blk, 0, stream>>>(data, len, d_answer.data());
  h_answer = d_answer.value(stream);
  return h_answer;
}

template <typename T>
__global__ void are_equal_kernel(T* embedding1, T* embedding2, size_t len, double* diff)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  if (embedding1[tid] != embedding2[tid]) {
    atomicAdd(diff, abs(embedding1[tid] - embedding2[tid]));
  }
}

template <typename T>
bool are_equal(T* embedding1, T* embedding2, size_t len, hipStream_t stream)
{
  double h_answer = 0.;
  rmm::device_scalar<double> d_answer(stream);
  raft::update_device(d_answer.data(), &h_answer, 1, stream);
  are_equal_kernel<<<raft::ceildiv(len, (size_t)32), 32, 0, stream>>>(
    embedding1, embedding2, len, d_answer.data());
  h_answer = d_answer.value(stream);

  double tolerance = 1.0;
  if (h_answer > tolerance) {
    std::cout << "Not equal, difference : " << h_answer << std::endl;
    return false;
  }
  return true;
}

class UMAPParametrizableTest : public ::testing::Test {
 protected:
  struct TestParams {
    bool fit_transform;
    bool supervised;
    bool knn_params;
    bool refine;
    int n_samples;
    int n_features;
    int n_clusters;
    double min_trustworthiness;
  };

  void get_embedding(raft::handle_t& handle,
                     float* X,
                     float* y,
                     float* embedding_ptr,
                     TestParams& test_params,
                     UMAPParams& umap_params)
  {
    hipStream_t stream = handle.get_stream();
    int& n_samples      = test_params.n_samples;
    int& n_features     = test_params.n_features;

    rmm::device_uvector<int64_t>* knn_indices_b{};
    rmm::device_uvector<float>* knn_dists_b{};
    int64_t* knn_indices{};
    float* knn_dists{};
    if (test_params.knn_params) {
      knn_indices_b = new rmm::device_uvector<int64_t>(n_samples * umap_params.n_neighbors, stream);
      knn_dists_b   = new rmm::device_uvector<float>(n_samples * umap_params.n_neighbors, stream);
      knn_indices   = knn_indices_b->data();
      knn_dists     = knn_dists_b->data();

      std::vector<float*> ptrs(1);
      std::vector<int> sizes(1);
      ptrs[0]  = X;
      sizes[0] = n_samples;

      raft::spatial::knn::brute_force_knn(handle,
                                          ptrs,
                                          sizes,
                                          n_features,
                                          X,
                                          n_samples,
                                          knn_indices,
                                          knn_dists,
                                          umap_params.n_neighbors);

      handle.sync_stream(stream);
    }

    float* model_embedding = nullptr;
    rmm::device_uvector<float>* model_embedding_b{};
    if (test_params.fit_transform) {
      model_embedding = embedding_ptr;
    } else {
      model_embedding_b =
        new rmm::device_uvector<float>(n_samples * umap_params.n_components, stream);
      model_embedding = model_embedding_b->data();
    }

    RAFT_CUDA_TRY(hipMemsetAsync(
      model_embedding, 0, n_samples * umap_params.n_components * sizeof(float), stream));

    handle.sync_stream(stream);

    if (test_params.supervised) {
      ML::UMAP::fit(
        handle, X, y, n_samples, n_features, knn_indices, knn_dists, &umap_params, model_embedding);
    } else {
      ML::UMAP::fit(handle,
                    X,
                    nullptr,
                    n_samples,
                    n_features,
                    knn_indices,
                    knn_dists,
                    &umap_params,
                    model_embedding);
    }

    if (test_params.refine) {
      std::cout << "using refine";
      if (test_params.supervised) {
        auto cgraph_coo = ML::UMAP::get_graph(handle, X, y, n_samples, n_features, &umap_params);
        ML::UMAP::refine(
          handle, X, n_samples, n_features, cgraph_coo.get(), &umap_params, model_embedding);
      } else {
        auto cgraph_coo =
          ML::UMAP::get_graph(handle, X, nullptr, n_samples, n_features, &umap_params);
        ML::UMAP::refine(
          handle, X, n_samples, n_features, cgraph_coo.get(), &umap_params, model_embedding);
      }
    }
    handle.sync_stream(stream);

    if (!test_params.fit_transform) {
      RAFT_CUDA_TRY(hipMemsetAsync(
        embedding_ptr, 0, n_samples * umap_params.n_components * sizeof(float), stream));

      handle.sync_stream(stream);

      ML::UMAP::transform(handle,
                          X,
                          n_samples,
                          umap_params.n_components,
                          knn_indices,
                          knn_dists,
                          X,
                          n_samples,
                          model_embedding,
                          n_samples,
                          &umap_params,
                          embedding_ptr);

      handle.sync_stream(stream);

      delete model_embedding_b;
    }

    if (test_params.knn_params) {
      delete knn_indices_b;
      delete knn_dists_b;
    }
  }

  void assertions(raft::handle_t& handle,
                  float* X,
                  float* embedding_ptr,
                  TestParams& test_params,
                  UMAPParams& umap_params)
  {
    hipStream_t stream = handle.get_stream();
    int& n_samples      = test_params.n_samples;
    int& n_features     = test_params.n_features;

    ASSERT_TRUE(!has_nan(embedding_ptr, n_samples * umap_params.n_components, stream));

    double trustworthiness =
      trustworthiness_score<float, raft::distance::DistanceType::L2SqrtUnexpanded>(
        handle,
        X,
        embedding_ptr,
        n_samples,
        n_features,
        umap_params.n_components,
        umap_params.n_neighbors);

    std::cout << "min. expected trustworthiness: " << test_params.min_trustworthiness << std::endl;
    std::cout << "trustworthiness: " << trustworthiness << std::endl;
    ASSERT_TRUE(trustworthiness > test_params.min_trustworthiness);
  }

  void test(TestParams& test_params, UMAPParams& umap_params)
  {
    std::cout << "\numap_params : [" << std::boolalpha << umap_params.n_neighbors << "-"
              << umap_params.n_components << "-" << umap_params.n_epochs << "-"
              << umap_params.random_state << std::endl;

    std::cout << "test_params : [" << std::boolalpha << test_params.fit_transform << "-"
              << test_params.supervised << "-" << test_params.refine << "-"
              << test_params.knn_params << "-" << test_params.n_samples << "-"
              << test_params.n_features << "-" << test_params.n_clusters << "-"
              << test_params.min_trustworthiness << "]" << std::endl;

    raft::handle_t handle;
    hipStream_t stream = handle.get_stream();
    int& n_samples      = test_params.n_samples;
    int& n_features     = test_params.n_features;

    UMAP::find_ab(handle, &umap_params);

    rmm::device_uvector<float> X_d(n_samples * n_features, stream);
    rmm::device_uvector<int> y_d(n_samples, stream);

    ML::Datasets::make_blobs(handle,
                             X_d.data(),
                             y_d.data(),
                             n_samples,
                             n_features,
                             test_params.n_clusters,
                             true,
                             nullptr,
                             nullptr,
                             1.f,
                             true,
                             -10.f,
                             10.f,
                             1234ULL);

    handle.sync_stream(stream);

    raft::linalg::convert_array((float*)y_d.data(), y_d.data(), n_samples, stream);

    handle.sync_stream(stream);

    rmm::device_uvector<float> embeddings1(n_samples * umap_params.n_components, stream);

    float* e1 = embeddings1.data();

#if CUDART_VERSION >= 11020
    // Always use random init w/ CUDA 11.2. For some reason the
    // spectral solver doesn't always converge w/ this CUDA version.
    umap_params.init         = 0;
    umap_params.random_state = 43;
    umap_params.n_epochs     = 500;
#endif
    get_embedding(handle, X_d.data(), (float*)y_d.data(), e1, test_params, umap_params);

    assertions(handle, X_d.data(), e1, test_params, umap_params);

    // v21.08: Reproducibility looks to be busted for CTK 11.4. Need to figure out
    // why this is happening and re-enable this.
#if CUDART_VERSION == 11040
    return;
#else
    // Disable reproducibility tests after transformation
    if (!test_params.fit_transform) { return; }
#endif

    rmm::device_uvector<float> embeddings2(n_samples * umap_params.n_components, stream);
    float* e2 = embeddings2.data();
    get_embedding(handle, X_d.data(), (float*)y_d.data(), e2, test_params, umap_params);

#if CUDART_VERSION >= 11020
    auto equal = are_equal(e1, e2, n_samples * umap_params.n_components, stream);

    if (!equal) {
      raft::print_device_vector("e1", e1, 25, std::cout);
      raft::print_device_vector("e2", e2, 25, std::cout);
    }

    ASSERT_TRUE(equal);
#else
    ASSERT_TRUE(
      raft::devArrMatch(e1, e2, n_samples * umap_params.n_components, raft::Compare<float>{}));
#endif
  }

  void SetUp() override
  {
    std::vector<TestParams> test_params_vec = {{false, false, false, true, 2000, 50, 20, 0.45},
                                               {true, false, false, false, 2000, 50, 20, 0.45},
                                               {false, true, false, true, 2000, 50, 20, 0.45},
                                               {false, false, true, false, 2000, 50, 20, 0.45},
                                               {true, true, false, true, 2000, 50, 20, 0.45},
                                               {true, false, true, false, 2000, 50, 20, 0.45},
                                               {false, true, true, true, 2000, 50, 20, 0.45},
                                               {true, true, true, false, 2000, 50, 20, 0.45}};

    std::vector<UMAPParams> umap_params_vec(4);
    umap_params_vec[0].n_components = 2;

    umap_params_vec[1].n_components = 10;

    umap_params_vec[2].n_components = 21;
    umap_params_vec[2].random_state = 43;
    umap_params_vec[2].init         = 0;
    umap_params_vec[2].n_epochs     = 500;

    umap_params_vec[3].n_components = 25;
    umap_params_vec[3].random_state = 43;
    umap_params_vec[3].init         = 0;
    umap_params_vec[3].n_epochs     = 500;

    for (auto& umap_params : umap_params_vec) {
      for (auto& test_params : test_params_vec) {
        test(test_params, umap_params);
      }
    }
  }

  void TearDown() override {}
};

typedef UMAPParametrizableTest UMAPParametrizableTest;
TEST_F(UMAPParametrizableTest, Result) {}
