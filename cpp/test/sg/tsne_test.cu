#ifndef IF_DEBUG
#define IF_DEBUG 1
#endif

#include <gtest/gtest.h>
#include "tsne/digits.h"
#include "tsne/tsne.cu"
//#include "tsne/Ground_Truth_TSNE.h"
#include <metrics/trustworthiness.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>

#include "cuda_utils.h"

using namespace MLCommon;
using namespace ML::Metrics;
using namespace MLCommon::Distance;

using namespace ML;

class TSNETest : public ::testing::Test {
 protected:
  void basicTest() {
    cumlHandle handle;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    float *X_d, *Y_d;
    MLCommon::allocate(X_d, n * p);
    MLCommon::allocate(Y_d, n * 2);
    MLCommon::updateDevice(X_d, digits.data(), n * p, stream);

    std::cout << "[>>>>]    Starting TSNE....\n";
    TSNE(handle, X_d, Y_d, n, p);
    std::cout << "[>>>>]    Got embeddings!....\n";

    std::cout << MLCommon::arr2Str(Y_d, 20, "embeddings", stream) << std::endl;

    std::cout << "Updating host" << std::endl;
    float embeddings_h[n * 2];
    MLCommon::updateHost(embeddings_h, Y_d, n * 2, stream);

    std::cout << "DONE!" << std::endl;

    CUDA_CHECK(hipPeekAtLastError());

    // Test trustworthiness
    // euclidean test
    score = trustworthiness_score<float, EucUnexpandedL2Sqrt>(handle, X_d, Y_d,
                                                              n, p, 2, 90);

    std::cout << "SCORE: " << score << std::endl;

    CUDA_CHECK(hipFree(Y_d));
    CUDA_CHECK(hipFree(X_d));

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {}

 protected:
  int n = 1797;
  int p = 64;
  double score;
};

typedef TSNETest TSNETestF;
TEST_F(TSNETestF, Result) { ASSERT_TRUE(0.9374 < score && score < 0.9376); }
