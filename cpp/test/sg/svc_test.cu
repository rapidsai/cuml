#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <test_utils.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <string>
#include <vector>
#include "common/cumlHandle.hpp"
#include "linalg/binary_op.h"
#include "linalg/map_then_reduce.h"
#include "linalg/transpose.h"
#include "matrix/grammatrix.h"
#include "matrix/kernelmatrices.h"
#include "random/make_blobs.h"
#include "random/rng.h"
#include "svm/smoblocksolve.h"
#include "svm/smosolver.h"
#include "svm/svc.hpp"
#include "svm/svm_model.h"
#include "svm/svm_parameter.h"
#include "svm/workingset.h"
#include "test_utils.h"

namespace ML {
namespace SVM {
using namespace MLCommon;
using namespace Matrix;

template <typename math_t>
class WorkingSetTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    allocate(f_dev, 10);
    allocate(y_dev, 10);
    allocate(alpha_dev, 10);
    updateDevice(f_dev, f_host, 10, stream);
    updateDevice(y_dev, y_host, 10, stream);
    updateDevice(alpha_dev, alpha_host, 10, stream);
  }

  void TearDown() override { CUDA_CHECK(hipStreamDestroy(stream)); }
  cumlHandle handle;
  hipStream_t stream;
  WorkingSet<math_t> *ws;

  math_t f_host[10] = {1, 3, 10, 4, 2, 8, 6, 5, 9, 7};
  math_t *f_dev;

  math_t y_host[10] = {-1, -1, -1, -1, -1, 1, 1, 1, 1, 1};
  math_t *y_dev;

  math_t C = 1.5;

  math_t alpha_host[10] = {0, 0, 0.1, 0.2, 1.5, 0, 0.2, 0.4, 1.5, 1.5};
  math_t *alpha_dev;  //   l  l  l/u  l/u    u  u  l/u  l/u  l    l

  int expected_idx[4] = {4, 3, 8, 2};
  int expected_idx2[4] = {8, 2, 4, 9};
};

typedef ::testing::Types<float, double> FloatTypes;

TYPED_TEST_CASE(WorkingSetTest, FloatTypes);

TYPED_TEST(WorkingSetTest, Init) {
  this->ws = new WorkingSet<TypeParam>(this->handle.getImpl(),
                                       this->handle.getStream(), 10);
  EXPECT_EQ(this->ws->GetSize(), 10);
  delete this->ws;

  this->ws =
    new WorkingSet<TypeParam>(this->handle.getImpl(), this->stream, 100000);
  EXPECT_EQ(this->ws->GetSize(), 1024);
  delete this->ws;
}

TYPED_TEST(WorkingSetTest, Select) {
  this->ws =
    new WorkingSet<TypeParam>(this->handle.getImpl(), this->stream, 10, 4);
  EXPECT_EQ(this->ws->GetSize(), 4);
  this->ws->SimpleSelect(this->f_dev, this->alpha_dev, this->y_dev, this->C);
  ASSERT_TRUE(devArrMatchHost(this->expected_idx, this->ws->GetIndices(),
                              this->ws->GetSize(), Compare<int>()));

  this->ws->Select(this->f_dev, this->alpha_dev, this->y_dev, this->C);
  ASSERT_TRUE(devArrMatchHost(this->expected_idx, this->ws->GetIndices(),
                              this->ws->GetSize(), Compare<int>()));
  this->ws->Select(this->f_dev, this->alpha_dev, this->y_dev, this->C);

  ASSERT_TRUE(devArrMatchHost(this->expected_idx2, this->ws->GetIndices(),
                              this->ws->GetSize(), Compare<int>()));
  delete this->ws;
}

//TYPED_TEST(WorkingSetTest, Priority) {
// See Issue #946
//}

template <typename math_t>
class KernelCacheTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    cublas_handle = handle.getImpl().getCublasHandle();
    allocate(x_dev, n_rows * n_cols);
    updateDevice(x_dev, x_host, n_rows * n_cols, stream);

    allocate(ws_idx_dev, n_ws);
    updateDevice(ws_idx_dev, ws_idx_host, n_ws, stream);
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
  }

  // Naive host side kernel implementation used for comparison
  void ApplyNonlin(Matrix::KernelParams params) {
    switch (params.kernel) {
      case Matrix::LINEAR:
        break;
      case Matrix::POLYNOMIAL:
        for (int z = 0; z < n_rows * n_ws; z++) {
          math_t val = params.gamma * tile_host_expected[z] + params.coef0;
          tile_host_expected[z] = pow(val, params.degree);
        }
        break;
      case Matrix::TANH:
        for (int z = 0; z < n_rows * n_ws; z++) {
          math_t val = params.gamma * tile_host_expected[z] + params.coef0;
          tile_host_expected[z] = tanh(val);
        }
        break;
      case Matrix::RBF:
        for (int i = 0; i < n_ws; i++) {
          for (int j = 0; j < n_rows; j++) {
            math_t d = 0;
            for (int k = 0; k < n_cols; k++) {
              int idx_i = ws_idx_host[i];
              math_t diff = x_host[idx_i + k * n_rows] - x_host[j + k * n_rows];
              d += diff * diff;
            }
            tile_host_expected[i * n_rows + j] = exp(-params.gamma * d);
          }
        }
        break;
    }
  }
  cumlHandle handle;
  hipblasHandle_t cublas_handle;
  hipStream_t stream;

  int n_rows = 4;
  int n_cols = 2;
  int n_ws = 3;

  math_t *x_dev;
  int *ws_idx_dev;

  math_t x_host[8] = {1, 2, 3, 4, 5, 6, 7, 8};
  int ws_idx_host[4] = {0, 1, 3};
  math_t tile_host_expected[12] = {26, 32, 38, 44, 32, 40,
                                   48, 56, 44, 56, 68, 80};
};

TYPED_TEST_CASE_P(KernelCacheTest);

TYPED_TEST_P(KernelCacheTest, EvalTest) {
  std::vector<Matrix::KernelParams> param_vec{
    Matrix::KernelParams{Matrix::LINEAR, 3, 1, 0},
    Matrix::KernelParams{Matrix::POLYNOMIAL, 2, 1.3, 1},
    Matrix::KernelParams{Matrix::TANH, 2, 0.5, 2.4},
    Matrix::KernelParams{Matrix::RBF, 2, 0.5, 0}};
  for (auto params : param_vec) {
    Matrix::GramMatrixBase<TypeParam> *kernel =
      Matrix::KernelFactory<TypeParam>::create(
        params, this->handle.getImpl().getCublasHandle());
    KernelCache<TypeParam> cache(this->handle.getImpl(), this->x_dev,
                                 this->n_rows, this->n_cols, this->n_ws,
                                 kernel);
    TypeParam *tile_dev = cache.GetTile(this->ws_idx_dev);
    // apply nonlinearity on tile_host_expected
    this->ApplyNonlin(params);
    ASSERT_TRUE(devArrMatchHost(this->tile_host_expected, tile_dev,
                                this->n_rows * this->n_ws,
                                CompareApprox<TypeParam>(1e-6f)));
    delete kernel;
  }
}

REGISTER_TYPED_TEST_CASE_P(KernelCacheTest, EvalTest);
INSTANTIATE_TYPED_TEST_CASE_P(My, KernelCacheTest, FloatTypes);

template <typename math_t>
class GetResultsTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
  }

  void TearDown() override { CUDA_CHECK(hipStreamDestroy(stream)); }

  void TestResults() {
    auto allocator = handle.getImpl().getDeviceAllocator();
    device_buffer<math_t> x_dev(allocator, stream, n_rows * n_cols);
    updateDevice(x_dev.data(), x_host, n_rows * n_cols, stream);
    device_buffer<math_t> f_dev(allocator, stream, n_rows);
    updateDevice(f_dev.data(), f_host, n_rows, stream);
    device_buffer<math_t> y_dev(allocator, stream, n_rows);
    updateDevice(y_dev.data(), y_host, n_rows, stream);
    device_buffer<math_t> alpha_dev(allocator, stream, n_rows);
    updateDevice(alpha_dev.data(), alpha_host, n_rows, stream);

    Results<math_t> res(handle.getImpl(), x_dev.data(), y_dev.data(), n_rows,
                        n_cols, C);
    res.Get(alpha_dev.data(), f_dev.data(), &dual_coefs, &n_coefs, &idx,
            &x_support, &b);

    ASSERT_EQ(n_coefs, 7);

    math_t dual_coefs_exp[] = {-0.1, -0.2, -1.5, 0.2, 0.4, 1.5, 1.5};
    EXPECT_TRUE(devArrMatchHost(dual_coefs_exp, dual_coefs, n_coefs,
                                CompareApprox<math_t>(1e-6f)));

    int idx_exp[] = {2, 3, 4, 6, 7, 8, 9};
    EXPECT_TRUE(devArrMatchHost(idx_exp, idx, n_coefs, Compare<int>()));

    math_t x_support_exp[] = {3, 4, 5, 7, 8, 9, 10, 13, 14, 15, 17, 18, 19, 20};
    EXPECT_TRUE(devArrMatchHost(x_support_exp, x_support, n_coefs * n_cols,
                                CompareApprox<math_t>(1e-6f)));

    EXPECT_FLOAT_EQ(b, -6.25f);

    if (n_coefs > 0) {
      allocator->deallocate(dual_coefs, n_coefs * sizeof(math_t), stream);
      allocator->deallocate(idx, n_coefs * sizeof(int), stream);
      allocator->deallocate(x_support, n_coefs * n_cols * sizeof(math_t),
                            stream);
    }

    // Modify the test by setting all SVs bound, then b is calculated differently
    math_t alpha_host2[10] = {0, 0, 1.5, 1.5, 1.5, 0, 1.5, 1.5, 1.5, 1.5};
    updateDevice(alpha_dev.data(), alpha_host2, n_rows, stream);
    res.Get(alpha_dev.data(), f_dev.data(), &dual_coefs, &n_coefs, &idx,
            &x_support, &b);
    EXPECT_FLOAT_EQ(b, -5.5f);
  }
  int n_rows = 10;
  int n_cols = 2;
  math_t x_host[20] = {1,  2,  3,  4,  5,  6,  7,  8,  9,  10,
                       11, 12, 13, 14, 15, 16, 17, 18, 19, 20};
  math_t f_host[10] = {1, 3, 10, 4, 2, 8, 6, 5, 9, 7};
  math_t y_host[10] = {-1, -1, -1, -1, -1, 1, 1, 1, 1, 1};
  math_t alpha_host[10] = {0, 0, 0.1, 0.2, 1.5, 0, 0.2, 0.4, 1.5, 1.5};
  //                      l  l  l/u  l/u    u  u  l/u  l/u  l    l
  math_t C = 1.5;

  math_t *dual_coefs;
  int n_coefs;
  int *idx;
  math_t *x_support;
  math_t b;

  cumlHandle handle;
  hipStream_t stream;
};

TYPED_TEST_CASE(GetResultsTest, FloatTypes);

TYPED_TEST(GetResultsTest, Results) { this->TestResults(); }

template <typename math_t>
class SmoUpdateTest : public ::testing::Test {
 protected:
  void SetUp() override {
    stream = handle.getImpl().getInternalStream(0);
    hipblasHandle_t cublas_handle = handle.getImpl().getCublasHandle();
    allocate(f_dev, n_rows, true);
    allocate(kernel_dev, n_rows * n_ws);
    updateDevice(kernel_dev, kernel_host, n_ws * n_rows, stream);
    allocate(delta_alpha_dev, n_ws);
    updateDevice(delta_alpha_dev, delta_alpha_host, n_ws, stream);
  }
  void RunTest() {
    SmoSolver<float> smo(handle.getImpl(), 1, 0.001, nullptr);
    smo.UpdateF(f_dev, n_rows, delta_alpha_dev, n_ws, kernel_dev);

    float f_host_expected[] = {0.1f, 7.4505806e-9f, 0.3f, 0.2f, 0.5f, 0.4f};
    devArrMatchHost(f_host_expected, f_dev, n_rows,
                    CompareApprox<math_t>(1e-6));
  }
  void TearDown() override {
    CUDA_CHECK(hipFree(delta_alpha_dev));
    CUDA_CHECK(hipFree(kernel_dev));
    CUDA_CHECK(hipFree(f_dev));
  }
  cumlHandle handle;
  hipStream_t stream;
  int n_rows = 6;
  int n_ws = 2;
  float *kernel_dev;
  float *f_dev;
  float *delta_alpha_dev;
  float kernel_host[12] = {3, 5, 4, 6, 5, 7, 4, 5, 7, 8, 10, 11};
  float delta_alpha_host[2] = {-0.1f, 0.1f};
};

TYPED_TEST_CASE(SmoUpdateTest, FloatTypes);
TYPED_TEST(SmoUpdateTest, Update) { this->RunTest(); }

template <typename math_t>
class SmoBlockSolverTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    cublas_handle = handle.getImpl().getCublasHandle();
    allocate(ws_idx_dev, n_ws);
    allocate(y_dev, n_rows);
    allocate(f_dev, n_rows);
    allocate(alpha_dev, n_rows, true);
    allocate(delta_alpha_dev, n_ws, true);
    allocate(kernel_dev, n_ws * n_rows);
    allocate(return_buff_dev, 2);

    updateDevice(ws_idx_dev, ws_idx_host, n_ws, stream);
    updateDevice(y_dev, y_host, n_rows, stream);
    updateDevice(f_dev, f_host, n_rows, stream);
    updateDevice(kernel_dev, kernel_host, n_ws * n_rows, stream);
  }

 public:  // because of the device lambda
  void testBlockSolve() {
    SmoBlockSolve<math_t, 1024><<<1, n_ws, 0, stream>>>(
      y_dev, n_rows, alpha_dev, n_ws, delta_alpha_dev, f_dev, kernel_dev,
      ws_idx_dev, 1.5f, 1e-3f, return_buff_dev, 1);
    CUDA_CHECK(hipPeekAtLastError());

    math_t return_buff_exp[2] = {0.2, 1};
    devArrMatchHost(return_buff_exp, return_buff_dev, 2,
                    CompareApprox<math_t>(1e-6));

    math_t *delta_alpha_calc;
    allocate(delta_alpha_calc, n_rows);
    LinAlg::binaryOp(
      delta_alpha_calc, y_dev, alpha_dev, n_rows,
      [] __device__(math_t a, math_t b) { return a * b; }, stream);
    devArrMatch(delta_alpha_dev, delta_alpha_calc, n_rows,
                CompareApprox<math_t>(1e-6));
    CUDA_CHECK(hipFree(delta_alpha_calc));
    math_t alpha_expected[] = {0, 0.1f, 0.1f, 0};
    devArrMatch(alpha_expected, alpha_dev, n_rows, CompareApprox<math_t>(1e-6));
  }

 protected:
  void TearDown() override {
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(y_dev));
    CUDA_CHECK(hipFree(f_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
    CUDA_CHECK(hipFree(alpha_dev));
    CUDA_CHECK(hipFree(delta_alpha_dev));
    CUDA_CHECK(hipFree(kernel_dev));
    CUDA_CHECK(hipFree(return_buff_dev));
  }

  cumlHandle handle;
  hipStream_t stream;
  hipblasHandle_t cublas_handle;

  int n_rows = 4;
  int n_cols = 2;
  int n_ws = 4;

  int *ws_idx_dev;
  math_t *y_dev;
  math_t *f_dev;
  math_t *alpha_dev;
  math_t *delta_alpha_dev;
  math_t *kernel_dev;
  math_t *return_buff_dev;

  int ws_idx_host[4] = {0, 1, 2, 3};
  math_t y_host[4] = {1, 1, -1, -1};
  math_t f_host[4] = {0.4, 0.3, 0.5, 0.1};
  math_t kernel_host[16] = {26, 32, 38, 44, 32, 40, 48, 56,
                            38, 48, 58, 68, 44, 56, 68, 80};
};

TYPED_TEST_CASE(SmoBlockSolverTest, FloatTypes);

// test a single iteration of the block solver
TYPED_TEST(SmoBlockSolverTest, SolveSingleTest) { this->testBlockSolve(); }

template <typename math_t>
struct smoInput {
  math_t C;
  math_t tol;
  KernelParams kernel_params;
  int max_iter;
  int max_inner_iter;
};

template <typename math_t>
struct svcInput {
  math_t C;
  math_t tol;
  KernelParams kernel_params;
  int n_rows;
  int n_cols;
  math_t *x_dev;
  math_t *y_dev;
  bool predict;
};

template <typename math_t>
struct smoOutput {
  int n_support;
  std::vector<math_t> dual_coefs;
  math_t b;
  std::vector<math_t> w;
  std::vector<math_t> x_support;
  std::vector<int> idx;
};

template <typename math_t>
struct svmTol {
  math_t b;
  math_t cs;
  int n_sv;
};

template <typename math_t>
class SmoSolverTest : public ::testing::Test {
 protected:
  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.setStream(stream);
    allocate(x_dev, n_rows * n_cols);
    allocate(ws_idx_dev, n_ws);
    allocate(y_dev, n_rows);
    allocate(y_pred, n_rows);
    allocate(f_dev, n_rows);
    allocate(alpha_dev, n_rows, true);
    allocate(delta_alpha_dev, n_ws, true);
    allocate(kernel_dev, n_ws * n_rows);
    allocate(return_buff_dev, 2);

    cublas_handle = handle.getImpl().getCublasHandle();

    updateDevice(x_dev, x_host, n_rows * n_cols, stream);
    updateDevice(ws_idx_dev, ws_idx_host, n_ws, stream);
    updateDevice(y_dev, y_host, n_rows, stream);
    updateDevice(f_dev, f_host, n_rows, stream);
    updateDevice(kernel_dev, kernel_host, n_ws * n_rows, stream);

    kernel = new Matrix::GramMatrixBase<math_t>(cublas_handle);
  }

  void FreeResultBuffers() {
    if (dual_coefs_d) CUDA_CHECK(hipFree(dual_coefs_d));
    if (idx_d) CUDA_CHECK(hipFree(idx_d));
    if (x_support_d) CUDA_CHECK(hipFree(x_support_d));
    dual_coefs_d = nullptr;
    idx_d = nullptr;
    x_support_d = nullptr;
  }
  void TearDown() override {
    delete kernel;
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(y_dev));
    CUDA_CHECK(hipFree(y_pred));
    CUDA_CHECK(hipFree(f_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
    CUDA_CHECK(hipFree(alpha_dev));
    CUDA_CHECK(hipFree(delta_alpha_dev));
    CUDA_CHECK(hipFree(kernel_dev));
    CUDA_CHECK(hipFree(return_buff_dev));
    FreeResultBuffers();
  }

 public:
  void blockSolveTest() {
    SmoBlockSolve<math_t, 1024><<<1, n_ws, 0, stream>>>(
      y_dev, n_rows, alpha_dev, n_ws, delta_alpha_dev, f_dev, kernel_dev,
      ws_idx_dev, 1.0, 1e-3, return_buff_dev);
    CUDA_CHECK(hipPeekAtLastError());

    math_t return_buff[2];
    updateHost(return_buff, return_buff_dev, 2, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    EXPECT_FLOAT_EQ(return_buff[0], 2.0f) << return_buff[0];
    EXPECT_LT(return_buff[1], 100) << return_buff[1];

    // check results won't work, because it expects that GetResults was called
    math_t *delta_alpha_calc;
    allocate(delta_alpha_calc, n_rows);
    LinAlg::binaryOp(
      delta_alpha_calc, y_dev, alpha_dev, n_rows,
      [] __device__(math_t a, math_t b) { return a * b; }, stream);
    devArrMatch(delta_alpha_dev, delta_alpha_calc, n_rows,
                CompareApprox<math_t>(1e-6));
    CUDA_CHECK(hipFree(delta_alpha_calc));

    math_t alpha_expected[] = {0.6f, 0, 1, 1, 0, 0.6f};
    //for C=10: {0.25f, 0, 2.25f, 3.75f, 0, 1.75f};
    devArrMatch(alpha_expected, alpha_dev, n_rows, CompareApprox<math_t>(1e-6));

    math_t host_alpha[6];
    updateHost(host_alpha, alpha_dev, n_rows, stream);

    math_t w[] = {0, 0};
    math_t ay = 0;
    for (int i = 0; i < n_rows; i++) {
      EXPECT_FLOAT_EQ(host_alpha[i], alpha_expected[i]) << "alpha " << i;
      w[0] += x_host[i] * host_alpha[i] * y_host[i];
      w[1] += x_host[i + n_rows] * host_alpha[i] * y_host[i];
      ay += host_alpha[i] * y_host[i];
    }
    EXPECT_FLOAT_EQ(ay, 0.0);
    EXPECT_FLOAT_EQ(w[0], -0.4);
    EXPECT_FLOAT_EQ(w[1], 1.2);
    // for C=10
    //EXPECT_FLOAT_EQ(w[0], -2.0);
    //EXPECT_FLOAT_EQ(w[1],  2.0);
  }

 protected:
  void checkResults(int n_coefs_exp, math_t *dual_coefs_exp, math_t b_exp,
                    math_t *w_exp, math_t *x_support_exp, int *idx_exp,
                    int n_coefs, int n_cols, math_t *dual_coefs_d = nullptr,
                    math_t b = 0, math_t *x_support_d = nullptr,
                    int *idx_d = nullptr, math_t b_tol = 0.001,
                    math_t cs_tol = 0.99999, int n_sv_diff = -1,
                    math_t ay_tol = 1e-5f) {
    if (n_sv_diff == -1) {
      n_sv_diff = n_coefs_exp * 0.01;
      if (n_coefs_exp > 10 && n_sv_diff < 3) n_sv_diff = 3;
    }
    EXPECT_LE(abs(n_coefs - n_coefs_exp), n_sv_diff);
    if (dual_coefs_exp) {
      EXPECT_TRUE(devArrMatchHost(dual_coefs_exp, dual_coefs_d, n_coefs,
                                  CompareApprox<math_t>(1e-3f)));
    }
    math_t *dual_coefs_host = new math_t[n_coefs];
    updateHost(dual_coefs_host, dual_coefs_d, n_coefs, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    math_t ay = 0;
    for (int i = 0; i < n_coefs; i++) {
      ay += dual_coefs_host[i];
    }
    // Test if \sum \alpha_i y_i = 0
    EXPECT_LT(abs(ay), ay_tol);

    if (x_support_exp) {
      EXPECT_TRUE(devArrMatchHost(x_support_exp, x_support_d, n_coefs * n_cols,
                                  CompareApprox<math_t>(1e-6f)));
    }

    if (idx_exp) {
      EXPECT_TRUE(devArrMatchHost(idx_exp, idx_d, n_coefs, Compare<int>()));
    }

    math_t *x_support_host = new math_t[n_coefs * n_cols];
    updateHost(x_support_host, x_support_d, n_coefs * n_cols, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (w_exp) {
      std::vector<math_t> w(n_cols, 0);
      for (int i = 0; i < n_coefs; i++) {
        for (int j = 0; j < n_cols; j++)
          w[j] += x_support_host[i + n_coefs * j] * dual_coefs_host[i];
      }
      // Calculate the cosine similarity between w and w_exp
      math_t abs_w = 0;
      math_t abs_w_exp = 0;
      math_t cs = 0;
      for (int i = 0; i < n_cols; i++) {
        abs_w += w[i] * w[i];
        abs_w_exp += w_exp[i] * w_exp[i];
        cs += w[i] * w_exp[i];
      }
      cs /= sqrt(abs_w * abs_w_exp);
      EXPECT_GT(cs, cs_tol);
    }

    EXPECT_LT(abs(b - b_exp), b_tol);

    delete[] dual_coefs_host;
    delete[] x_support_host;
  }

  void checkResults(svmModel<math_t> model, smoOutput<math_t> expected,
                    svmTol<math_t> tol = svmTol<math_t>{0.001, 0.99999, -1}) {
    math_t *dcoef_exp =
      expected.dual_coefs.size() > 0 ? expected.dual_coefs.data() : nullptr;
    math_t *w_exp = expected.w.size() > 0 ? expected.w.data() : nullptr;
    math_t *x_sv_exp =
      expected.x_support.size() > 0 ? expected.x_support.data() : nullptr;
    int *idx_exp = expected.idx.size() > 0 ? expected.idx.data() : nullptr;

    checkResults(expected.n_support, dcoef_exp, expected.b, w_exp, x_sv_exp,
                 idx_exp, model.n_support, model.n_cols, model.dual_coefs,
                 model.b, model.x_support, model.support_idx, tol.b, tol.cs,
                 tol.n_sv);
  }
  cumlHandle handle;
  hipStream_t stream;
  Matrix::GramMatrixBase<math_t> *kernel;
  int n_rows = 6;
  const int n_cols = 2;
  int n_ws = 6;

  math_t *x_dev;
  int *ws_idx_dev;
  math_t *y_dev;
  math_t *y_pred;
  math_t *f_dev;
  math_t *alpha_dev;
  math_t *delta_alpha_dev;
  math_t *kernel_dev;
  math_t *return_buff_dev;

  math_t x_host[12] = {1, 2, 1, 2, 1, 2, 1, 1, 2, 2, 3, 3};
  int ws_idx_host[6] = {0, 1, 2, 3, 4, 5};
  math_t y_host[6] = {-1, -1, 1, -1, 1, 1};

  math_t f_host[6] = {1, 1, -1, 1, -1, -1};

  math_t kernel_host[36] = {2, 3, 3, 4, 4,  5,  3, 5, 4, 6,  5,  7,
                            3, 4, 5, 6, 7,  8,  4, 6, 6, 8,  8,  10,
                            4, 5, 7, 8, 10, 11, 5, 7, 8, 10, 11, 13};
  hipblasHandle_t cublas_handle;

  math_t *dual_coefs_d = nullptr;
  int n_coefs;
  int *idx_d = nullptr;
  math_t *x_support_d = nullptr;
  math_t b;
};

TYPED_TEST_CASE(SmoSolverTest, FloatTypes);

TYPED_TEST(SmoSolverTest, BlockSolveTest) { this->blockSolveTest(); }

std::string kernelName(KernelParams k) {
  std::vector<std::string> names{"linear", "poly", "rbf", "tanh"};
  return names[k.kernel];
}

template <typename math_t>
std::ostream &operator<<(std::ostream &os, const smoInput<math_t> &b) {
  os << kernelName(b.kernel_params) << ", C=" << b.C << ", tol=" << b.tol;
  return os;
}

TYPED_TEST(SmoSolverTest, SmoSolveTest) {
  std::vector<std::pair<smoInput<TypeParam>, smoOutput<TypeParam>>> data{
    {smoInput<TypeParam>{1, 0.001, KernelParams{LINEAR, 3, 1, 0}, 100, 1},
     smoOutput<TypeParam>{4,                         // n_sv
                          {-0.6, 1, -1, 0.6},        // dual_coefs
                          -1.8,                      // b
                          {-0.4, 1.2},               // w
                          {1, 1, 2, 2, 1, 2, 2, 3},  // x_support
                          {0, 2, 3, 5}}},            // support idx
    {smoInput<TypeParam>{10, 0.001, KernelParams{LINEAR, 3, 1, 0}, 100, 1},
     smoOutput<TypeParam>{3, {-2, 4, -2, 0, 0}, -1.0, {-2, 2}, {}, {}}},
    {smoInput<TypeParam>{1, 1e-6, KernelParams{POLYNOMIAL, 3, 1, 1}, 100, 1},
     smoOutput<TypeParam>{3,
                          {-0.02556136, 0.03979708, -0.01423571},
                          -1.07739149,
                          {},
                          {1, 1, 2, 1, 2, 2},
                          {0, 2, 3}}}};

  for (auto d : data) {
    auto p = d.first;
    auto exp = d.second;
    SCOPED_TRACE(p);
    GramMatrixBase<TypeParam> *kernel = KernelFactory<TypeParam>::create(
      p.kernel_params, this->handle.getImpl().getCublasHandle());
    SmoSolver<TypeParam> smo(this->handle.getImpl(), p.C, p.tol, kernel);
    svmModel<TypeParam> model{0,       this->n_cols, 0, nullptr,
                              nullptr, nullptr,      0, nullptr};
    smo.Solve(this->x_dev, this->n_rows, this->n_cols, this->y_dev,
              &model.dual_coefs, &model.n_support, &model.x_support,
              &model.support_idx, &model.b, p.max_iter, p.max_inner_iter);
    this->checkResults(model, exp);
    svmFreeBuffers(this->handle, model);
  }
}

TYPED_TEST(SmoSolverTest, SvcTest) {
  std::vector<std::pair<svcInput<TypeParam>, smoOutput<TypeParam>>> data{
    {svcInput<TypeParam>{1, 0.001, KernelParams{LINEAR, 3, 1, 0}, this->n_rows,
                         this->n_cols, this->x_dev, this->y_dev, true},
     smoOutput<TypeParam>{4,
                          {-0.6, 1, -1, 0.6},
                          -1.8f,
                          {-0.4, 1.2},
                          {1, 1, 2, 2, 1, 2, 2, 3},
                          {0, 2, 3, 5}}},
    {svcInput<TypeParam>{1, 1e-6, KernelParams{POLYNOMIAL, 3, 1, 0},
                         this->n_rows, this->n_cols, this->x_dev, this->y_dev,
                         true},
     smoOutput<TypeParam>{3,
                          {-0.03900895, 0.05904058, -0.02003163},
                          -0.99999959,
                          {},
                          {1, 1, 2, 1, 2, 2},
                          {0, 2, 3}}},
    {svcInput<TypeParam>{10, 1e-6, KernelParams{TANH, 3, 0.3, 1.0},
                         this->n_rows, this->n_cols, this->x_dev, this->y_dev,
                         false},
     smoOutput<TypeParam>{6,
                          {-10., -10., 10., -10., 10., 10.},
                          -0.3927505,
                          {},
                          {1, 2, 1, 2, 1, 2, 1, 1, 2, 2, 3, 3},
                          {0, 1, 2, 3, 4, 5}}},
    {svcInput<TypeParam>{1, 1.0e-6, KernelParams{RBF, 0, 0.15, 0}, this->n_rows,
                         this->n_cols, this->x_dev, this->y_dev, true},
     smoOutput<TypeParam>{6,
                          {-1., -1, 1., -1., 1, 1.},
                          0,
                          {},
                          {1, 2, 1, 2, 1, 2, 1, 1, 2, 2, 3, 3},
                          {0, 1, 2, 3, 4, 5}}}};

  for (auto d : data) {
    auto p = d.first;
    auto exp = d.second;
    SCOPED_TRACE(kernelName(p.kernel_params));
    SVC<TypeParam> svc(this->handle, p.C, p.tol, p.kernel_params);
    svc.fit(p.x_dev, p.n_rows, p.n_cols, p.y_dev);
    this->checkResults(svc.model, exp);
    device_buffer<TypeParam> y_pred(this->handle.getDeviceAllocator(),
                                    this->stream, p.n_rows);
    if (p.predict) {
      svc.predict(p.x_dev, p.n_rows, p.n_cols, y_pred.data());
      EXPECT_TRUE(devArrMatch(this->y_dev, y_pred.data(), p.n_rows,
                              CompareApprox<TypeParam>(1e-6f)));
    }
  }
}

struct blobInput {
  double C;
  double tol;
  KernelParams kernel_params;
  int n_rows;
  int n_cols;
};

std::ostream &operator<<(std::ostream &os, const blobInput &b) {
  os << kernelName(b.kernel_params) << " " << b.n_rows << "x" << b.n_cols;
  return os;
}

// until there is progress with Issue #935
template <typename inType, typename outType>
__global__ void cast(outType *out, int n, inType *in) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < n) out[tid] = in[tid];
}

// To have the same input data for both single and double precision,
// we generate the blobs in single precision only, and cast to dp if needed.
template <typename math_t>
void make_blobs(math_t *x, math_t *y, int n_rows, int n_cols, int n_cluster,
                std::shared_ptr<MLCommon::deviceAllocator> &allocator,
                hipblasHandle_t hipblas.h, hipStream_t stream,
                float *centers = nullptr) {
  device_buffer<float> x_float(allocator, stream, n_rows * n_cols);
  device_buffer<int> y_int(allocator, stream, n_rows);

  Random::make_blobs(x_float.data(), y_int.data(), n_rows, n_cols, n_cluster,
                     allocator, stream, centers, (float *)nullptr, 1.0f, true,
                     -2.0f, 2.0f, 0);
  int TPB = 256;
  if (std::is_same<float, math_t>::value) {
    LinAlg::transpose(x_float.data(), (float *)x, n_cols, n_rows, hipblas.h,
                      stream);
  } else {
    device_buffer<math_t> x2(allocator, stream, n_rows * n_cols);
    cast<<<MLCommon::ceildiv(n_rows * n_cols, TPB), TPB, 0, stream>>>(
      x2.data(), n_rows * n_cols, x_float.data());
    LinAlg::transpose(x2.data(), x, n_cols, n_rows, hipblas.h, stream);
    CUDA_CHECK(hipPeekAtLastError());
  }
  cast<<<MLCommon::ceildiv(n_rows, TPB), TPB, 0, stream>>>(y, n_rows,
                                                           y_int.data());
  CUDA_CHECK(hipPeekAtLastError());
}

TYPED_TEST(SmoSolverTest, Blobs) {
  std::vector<std::pair<blobInput, smoOutput<TypeParam>>> data{
    {blobInput{1, 0.001, KernelParams{LINEAR, 3, 1, 0}, 100, 1},
     smoOutput<TypeParam>{94, {}, 0.5694, {-0.59181675}, {}, {}}},
    {blobInput{1, 0.001, KernelParams{LINEAR, 3, 1, 0}, 2, 100},
     smoOutput<TypeParam>{2, {0.00377128, -0.00377128}, 0.0392874, {}, {}, {}}},
    {blobInput{1, 0.001, KernelParams{LINEAR, 3, 1, 0}, 100, 100},
     smoOutput<TypeParam>{
       21,
       {},
       -0.027839,
       {-0.01694475, -0.0058431,  -0.00540597, 0.03296302,  0.00389359,
        0.02106245,  0.0039061,   -0.01332385, 0.00660107,  0.02471894,
        -0.00449132, 0.0036424,   0.01026903,  0.02074523,  -0.01541529,
        0.00959903,  -0.01748434, 0.00014968,  0.00110548,  -0.00965647,
        0.00798169,  0.00621375,  0.01358159,  0.00528406,  0.01153438,
        -0.01368673, -0.03216547, -0.00200592, 0.0120133,   0.00176928,
        0.00642773,  0.01619464,  -0.00064005, 0.01400618,  0.01727131,
        -0.02331973, 0.00197682,  -0.00378402, 0.00612855,  -0.02184831,
        -0.00363246, -0.0127956,  0.02767534,  -0.00553812, -0.02225143,
        0.02767332,  0.00623573,  -0.02094788, -0.02012747, 0.01102353,
        0.00080433,  -0.0311776,  -0.01562613, -0.02279347, 0.02422357,
        -0.00057321, -0.02881205, 0.00038414,  0.00919522,  0.00644822,
        -0.02633642, -0.00157511, -0.00627405, -0.00604624, 0.0104084,
        0.00264318,  0.00155375,  -0.00997641, 0.00110094,  0.01547085,
        -0.00342524, -0.00329727, 0.00836803,  -0.00882695, -0.0022986,
        0.0046427,   -0.0214064,  -0.01128186, 0.00787328,  -0.02030728,
        -0.00835369, 0.0141688,   -0.01291866, 0.0155667,   -0.01748242,
        -0.01181509, -0.00194211, -0.01517817, 0.01449254,  -0.00437902,
        0.00218727,  0.01684742,  -0.00274239, 0.01743845,  0.02874734,
        0.00412968,  0.00070507,  -0.00334403, 0.02057458,  -0.01662802},
       {},
       {}}},
    {blobInput{1, 1e-3, KernelParams{LINEAR, 3, 1, 0}, 1000, 100},
     smoOutput<TypeParam>{
       34,
       {},
       0.0681913,
       {-2.74771248e-02, -1.93489987e-03, -5.10868053e-03, 3.62352833e-02,
        -4.95548876e-03, 2.12629716e-02,  7.24245748e-03,  -9.81239809e-03,
        1.04116603e-02,  1.98374788e-02,  -8.85246492e-03, 3.87150029e-03,
        1.33185355e-02,  2.48162036e-02,  -2.19338463e-02, -5.84521038e-03,
        -1.52917105e-02, -2.02794426e-03, 1.02024677e-02,  -9.34250325e-03,
        5.39213720e-03,  1.04247828e-02,  2.11718472e-02,  5.84453748e-05,
        9.20193459e-03,  -2.68006157e-02, -3.37565162e-02, 7.36789225e-03,
        4.61224812e-03,  -1.16150969e-03, 5.51388041e-03,  2.28353876e-02,
        1.01278434e-02,  8.88320327e-03,  2.84587444e-02,  -2.69580067e-02,
        1.28087948e-02,  3.91011504e-03,  5.37012676e-03,  -3.16363924e-02,
        -3.35284170e-03, -6.50114513e-03, 3.20651614e-02,  -1.16159910e-02,
        -1.38094305e-02, 3.07976442e-02,  1.63332093e-02,  -2.74646081e-02,
        -3.29649271e-02, 1.47660371e-02,  6.09094255e-03,  -3.94666490e-02,
        -1.23632624e-02, -3.69395768e-02, 2.86938112e-02,  -6.44606846e-03,
        -2.28087983e-02, 7.52382321e-03,  1.09257772e-02,  1.74754686e-03,
        -2.18251777e-02, 2.26055336e-03,  -7.57926813e-03, -8.66892829e-03,
        5.56296376e-03,  1.87841417e-03,  -1.16321532e-02, -3.61208310e-03,
        1.23176737e-02,  2.37739726e-02,  -1.27088417e-02, -5.08065174e-03,
        2.75792244e-03,  -6.36667336e-03, -4.09090428e-03, 5.91545951e-03,
        -1.50748294e-02, -1.32456566e-02, 5.02961559e-03,  -2.07146521e-02,
        1.53120474e-03,  1.16797660e-02,  -1.32767277e-02, 1.50207104e-02,
        -2.39799601e-02, -7.01546552e-03, 8.88361434e-04,  -1.60787453e-02,
        1.89898754e-02,  -4.73620907e-03, -4.03131155e-03, 2.93305320e-02,
        1.33827967e-03,  1.94703514e-02,  2.87304765e-02,  9.69693810e-03,
        5.91821346e-05,  -9.06411618e-03, 3.03503309e-02,  -1.82080580e-02},
       {},
       {}}},
    {blobInput{1, 1e-3, KernelParams{LINEAR, 3, 0.001, 0}, 100, 1000},
     smoOutput<TypeParam>{41, {}, -0.0077539393, {}, {}, {}}}};

  std::vector<svmTol<TypeParam>> tolerance{
    svmTol<TypeParam>{1e-3, 0.99999, -1}, svmTol<TypeParam>{1e-3, 0.99999, -1},
    svmTol<TypeParam>{1e-3, 0.99999, -1}, svmTol<TypeParam>{0.05, 0.95, 10},
    svmTol<TypeParam>{0.05, 0.95, 8}};
  int i = 0;
  auto allocator = this->handle.getDeviceAllocator();
  for (auto d : data) {
    auto p = d.first;
    SCOPED_TRACE(p);
    device_buffer<TypeParam> x(allocator, this->stream, p.n_rows * p.n_cols);
    device_buffer<TypeParam> y(allocator, this->stream, p.n_rows);
    make_blobs(x.data(), y.data(), p.n_rows, p.n_cols, 2, allocator,
               this->handle.getImpl().getCublasHandle(), this->stream);
    SVC<TypeParam> svc(this->handle, p.C, p.tol, p.kernel_params);
    svc.fit(x.data(), p.n_rows, p.n_cols, y.data());
    //std::cout << p << ": " << svc.model.n_support << " " << svc.model.b << "\n";
    auto exp = d.second;
    svmTol<TypeParam> tol = tolerance[i];
    i++;
    this->checkResults(svc.model, exp, tol);
    device_buffer<TypeParam> y_pred(this->handle.getDeviceAllocator(),
                                    this->stream, p.n_rows);
    svc.predict(x.data(), p.n_rows, p.n_cols, y_pred.data());
  }
}

struct is_same_functor {
  template <typename Tuple>
  __host__ __device__ int operator()(Tuple t) {
    return thrust::get<0>(t) == thrust::get<1>(t);
  }
};

TYPED_TEST(SmoSolverTest, BlobPredict) {
  // Pair.second is the expected accuracy. It might change if the Rng changes.
  std::vector<std::pair<blobInput, TypeParam>> data{
    {blobInput{1, 0.001, KernelParams{LINEAR, 3, 1, 0}, 200, 10}, 98},
    {blobInput{1, 0.001, KernelParams{POLYNOMIAL, 3, 1, 0}, 200, 10}, 98},
    {blobInput{1, 0.001, KernelParams{RBF, 3, 1, 0}, 200, 2}, 98},
    {blobInput{1, 0.009, KernelParams{TANH, 3, 0.1, 0}, 200, 10}, 98}};

  // This should be larger then N_PRED_BATCH in svcPredict
  const int n_pred = 5000;

  auto allocator = this->handle.getDeviceAllocator();

  for (auto d : data) {
    auto p = d.first;
    SCOPED_TRACE(p);
    // explicit centers for the blobs
    device_buffer<float> centers(allocator, this->stream, 2 * p.n_cols);
    thrust::device_ptr<float> thrust_ptr(centers.data());
    thrust::fill(thrust::cuda::par.on(this->stream), thrust_ptr,
                 thrust_ptr + p.n_cols, -5.0f);
    thrust::fill(thrust::cuda::par.on(this->stream), thrust_ptr + p.n_cols,
                 thrust_ptr + 2 * p.n_cols, +5.0f);

    device_buffer<TypeParam> x(allocator, this->stream, p.n_rows * p.n_cols);
    device_buffer<TypeParam> y(allocator, this->stream, p.n_rows);
    device_buffer<TypeParam> x_pred(allocator, this->stream, n_pred * p.n_cols);
    device_buffer<TypeParam> y_pred(allocator, this->stream, n_pred);

    make_blobs(x.data(), y.data(), p.n_rows, p.n_cols, 2, allocator,
               this->handle.getImpl().getCublasHandle(), this->stream,
               centers.data());
    SVC<TypeParam> svc(this->handle, p.C, p.tol, p.kernel_params);
    svc.fit(x.data(), p.n_rows, p.n_cols, y.data());

    // Create a different dataset for prediction
    make_blobs(x_pred.data(), y_pred.data(), n_pred, p.n_cols, 2, allocator,
               this->handle.getImpl().getCublasHandle(), this->stream,
               centers.data());
    device_buffer<TypeParam> y_pred2(this->handle.getDeviceAllocator(),
                                     this->stream, n_pred);
    svc.predict(x_pred.data(), n_pred, p.n_cols, y_pred2.data());

    // Count the number of correct predictions
    device_buffer<int> is_correct(this->handle.getDeviceAllocator(),
                                  this->stream, n_pred);
    thrust::device_ptr<TypeParam> ptr1(y_pred.data());
    thrust::device_ptr<TypeParam> ptr2(y_pred2.data());
    thrust::device_ptr<int> ptr3(is_correct.data());
    auto first = thrust::make_zip_iterator(thrust::make_tuple(ptr1, ptr2));
    auto last = thrust::make_zip_iterator(
      thrust::make_tuple(ptr1 + n_pred, ptr2 + n_pred));
    thrust::transform(thrust::cuda::par.on(this->stream), first, last, ptr3,
                      is_same_functor());
    int n_correct =
      thrust::reduce(thrust::cuda::par.on(this->stream), ptr3, ptr3 + n_pred);

    TypeParam accuracy = 100 * n_correct / n_pred;
    TypeParam accuracy_exp = d.second;
    EXPECT_GE(accuracy, accuracy_exp);
  }
}

TYPED_TEST(SmoSolverTest, MemoryLeak) {
  std::vector<std::pair<blobInput, smoOutput<TypeParam>>> data{
    {blobInput{1, 0.001, KernelParams{LINEAR, 3, 0.01, 0}, 1000, 1000},
     smoOutput<TypeParam>{34, {}, 0.0681913, {}, {}, {}}}};
  size_t free1, total, free2;
  CUDA_CHECK(hipMemGetInfo(&free1, &total));
  auto allocator = this->handle.getDeviceAllocator();
  for (auto d : data) {
    auto p = d.first;
    SCOPED_TRACE(p);

    device_buffer<TypeParam> x(allocator, this->stream, p.n_rows * p.n_cols);
    device_buffer<TypeParam> y(allocator, this->stream, p.n_rows);
    make_blobs(x.data(), y.data(), p.n_rows, p.n_cols, 2, allocator,
               this->handle.getImpl().getCublasHandle(), this->stream);

    SVC<TypeParam> svc(this->handle, p.C, p.tol, p.kernel_params);
    svc.fit(x.data(), p.n_rows, p.n_cols, y.data());
    device_buffer<TypeParam> y_pred(this->handle.getDeviceAllocator(),
                                    this->stream, p.n_rows);
    CUDA_CHECK(hipStreamSynchronize(this->stream));
    CUDA_CHECK(hipMemGetInfo(&free2, &total));
    float delta = (free1 - free2);
    EXPECT_GT(delta, p.n_rows * p.n_cols * 4);
    CUDA_CHECK(hipStreamSynchronize(this->stream));
    svc.predict(x.data(), p.n_rows, p.n_cols, y_pred.data());
  }
  CUDA_CHECK(hipMemGetInfo(&free2, &total));
  float delta = (free1 - free2);
  EXPECT_EQ(delta, 0);
}

};  // namespace SVM
};  // end namespace ML
