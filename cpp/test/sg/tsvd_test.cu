/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/decomposition/params.hpp>
#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/random/rng.hpp>
#include <test_utils.h>
#include <tsvd/tsvd.cuh>
#include <vector>

namespace ML {

template <typename T>
struct TsvdInputs {
  T tolerance;
  int n_row;
  int n_col;
  int n_row2;
  int n_col2;
  float redundancy;
  unsigned long long int seed;
  int algo;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const TsvdInputs<T>& dims)
{
  return os;
}

template <typename T>
class TsvdTest : public ::testing::TestWithParam<TsvdInputs<T>> {
 public:
  TsvdTest()
    : params(::testing::TestWithParam<TsvdInputs<T>>::GetParam()),
      stream(handle.get_stream()),
      components(0, stream),
      components_ref(0, stream),
      data2(0, stream),
      data2_back(0, stream)
  {
    basicTest();
    advancedTest();
  }

 protected:
  void basicTest()
  {
    raft::random::Rng r(params.seed, raft::random::GenPC);
    int len = params.n_row * params.n_col;

    rmm::device_uvector<T> data(len, stream);

    std::vector<T> data_h = {1.0, 2.0, 4.0, 2.0, 4.0, 5.0, 5.0, 4.0, 2.0, 1.0, 6.0, 4.0};
    data_h.resize(len);
    raft::update_device(data.data(), data_h.data(), len, stream);

    int len_comp = params.n_col * params.n_col;
    components.resize(len_comp, stream);
    rmm::device_uvector<T> singular_vals(params.n_col, stream);

    std::vector<T> components_ref_h = {
      -0.3951, 0.1532, 0.9058, -0.7111, -0.6752, -0.1959, -0.5816, 0.7215, -0.3757};
    components_ref_h.resize(len_comp);

    components_ref.resize(len_comp, stream);
    raft::update_device(components_ref.data(), components_ref_h.data(), len_comp, stream);

    paramsTSVD prms;
    prms.n_cols       = params.n_col;
    prms.n_rows       = params.n_row;
    prms.n_components = params.n_col;
    if (params.algo == 0)
      prms.algorithm = solver::COV_EIG_DQ;
    else
      prms.algorithm = solver::COV_EIG_JACOBI;

    tsvdFit(handle, data.data(), components.data(), singular_vals.data(), prms, stream);
  }

  void advancedTest()
  {
    raft::random::Rng r(params.seed, raft::random::GenPC);
    int len = params.n_row2 * params.n_col2;

    paramsTSVD prms;
    prms.n_cols       = params.n_col2;
    prms.n_rows       = params.n_row2;
    prms.n_components = params.n_col2;
    if (params.algo == 0)
      prms.algorithm = solver::COV_EIG_DQ;
    else if (params.algo == 1)
      prms.algorithm = solver::COV_EIG_JACOBI;
    else
      prms.n_components = params.n_col2 - 15;

    data2.resize(len, stream);
    int redundant_cols = int(params.redundancy * params.n_col2);
    int redundant_len  = params.n_row2 * redundant_cols;

    int informative_cols = params.n_col2 - redundant_cols;
    int informative_len  = params.n_row2 * informative_cols;

    r.uniform(data2.data(), informative_len, T(-1.0), T(1.0), stream);
    CUDA_CHECK(hipMemcpyAsync(data2.data() + informative_len,
                               data2.data(),
                               redundant_len * sizeof(T),
                               hipMemcpyDeviceToDevice,
                               stream));
    rmm::device_uvector<T> data2_trans(prms.n_rows * prms.n_components, stream);

    int len_comp = params.n_col2 * prms.n_components;
    rmm::device_uvector<T> components2(len_comp, stream);
    rmm::device_uvector<T> explained_vars2(prms.n_components, stream);
    rmm::device_uvector<T> explained_var_ratio2(prms.n_components, stream);
    rmm::device_uvector<T> singular_vals2(prms.n_components, stream);

    tsvdFitTransform(handle,
                     data2.data(),
                     data2_trans.data(),
                     components2.data(),
                     explained_vars2.data(),
                     explained_var_ratio2.data(),
                     singular_vals2.data(),
                     prms,
                     stream);

    data2_back.resize(len, stream);
    tsvdInverseTransform(
      handle, data2_trans.data(), components2.data(), data2_back.data(), prms, stream);
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream = 0;

  TsvdInputs<T> params;
  rmm::device_uvector<T> components, components_ref, data2, data2_back;
};

const std::vector<TsvdInputs<float>> inputsf2 = {{0.01f, 4, 3, 1024, 128, 0.25f, 1234ULL, 0},
                                                 {0.01f, 4, 3, 1024, 128, 0.25f, 1234ULL, 1},
                                                 {0.04f, 4, 3, 512, 64, 0.25f, 1234ULL, 2},
                                                 {0.04f, 4, 3, 512, 64, 0.25f, 1234ULL, 2}};

const std::vector<TsvdInputs<double>> inputsd2 = {{0.01, 4, 3, 1024, 128, 0.25f, 1234ULL, 0},
                                                  {0.01, 4, 3, 1024, 128, 0.25f, 1234ULL, 1},
                                                  {0.05, 4, 3, 512, 64, 0.25f, 1234ULL, 2},
                                                  {0.05, 4, 3, 512, 64, 0.25f, 1234ULL, 2}};

typedef TsvdTest<float> TsvdTestLeftVecF;
TEST_P(TsvdTestLeftVecF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(components.data(),
                                components_ref.data(),
                                (params.n_col * params.n_col),
                                raft::CompareApproxAbs<float>(params.tolerance),
                                handle.get_stream()));
}

typedef TsvdTest<double> TsvdTestLeftVecD;
TEST_P(TsvdTestLeftVecD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(components.data(),
                                components_ref.data(),
                                (params.n_col * params.n_col),
                                raft::CompareApproxAbs<double>(params.tolerance),
                                handle.get_stream()));
}

typedef TsvdTest<float> TsvdTestDataVecF;
TEST_P(TsvdTestDataVecF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(data2.data(),
                                data2_back.data(),
                                (params.n_col2 * params.n_col2),
                                raft::CompareApproxAbs<float>(params.tolerance),
                                handle.get_stream()));
}

typedef TsvdTest<double> TsvdTestDataVecD;
TEST_P(TsvdTestDataVecD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(data2.data(),
                                data2_back.data(),
                                (params.n_col2 * params.n_col2),
                                raft::CompareApproxAbs<double>(params.tolerance),
                                handle.get_stream()));
}

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestLeftVecF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestLeftVecD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestDataVecF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestDataVecD, ::testing::ValuesIn(inputsd2));

}  // end namespace ML
