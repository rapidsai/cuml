/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <test_utils.h>
#include <cuml/decomposition/params.hpp>
#include <raft/random/rng.cuh>
#include <tsvd/tsvd.cuh>
#include <vector>

namespace ML {

using namespace MLCommon;

template <typename T>
struct TsvdInputs {
  T tolerance;
  int len;
  int n_row;
  int n_col;
  int len2;
  int n_row2;
  int n_col2;
  unsigned long long int seed;
  int algo;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const TsvdInputs<T>& dims) {
  return os;
}

template <typename T>
class TsvdTest : public ::testing::TestWithParam<TsvdInputs<T>> {
 protected:
  void basicTest() {
    params = ::testing::TestWithParam<TsvdInputs<T>>::GetParam();
    raft::random::Rng r(params.seed, raft::random::GenTaps);
    int len = params.len;

    raft::allocate(data, len);

    std::vector<T> data_h = {1.0, 2.0, 4.0, 2.0, 4.0, 5.0,
                             5.0, 4.0, 2.0, 1.0, 6.0, 4.0};
    data_h.resize(len);
    raft::update_device(data, data_h.data(), len, stream);

    int len_comp = params.n_col * params.n_col;
    raft::allocate(components, len_comp);
    raft::allocate(singular_vals, params.n_col);

    std::vector<T> components_ref_h = {-0.3951, 0.1532,  0.9058,
                                       -0.7111, -0.6752, -0.1959,
                                       -0.5816, 0.7215,  -0.3757};
    components_ref_h.resize(len_comp);

    raft::allocate(components_ref, len_comp);
    raft::update_device(components_ref, components_ref_h.data(), len_comp,
                        stream);

    paramsTSVD prms;
    prms.n_cols = params.n_col;
    prms.n_rows = params.n_row;
    prms.n_components = params.n_col;
    if (params.algo == 0)
      prms.algorithm = solver::COV_EIG_DQ;
    else
      prms.algorithm = solver::COV_EIG_JACOBI;

    tsvdFit(handle, data, components, singular_vals, prms, stream);
  }

  void advancedTest() {
    params = ::testing::TestWithParam<TsvdInputs<T>>::GetParam();
    raft::random::Rng r(params.seed, raft::random::GenTaps);
    int len = params.len2;

    paramsTSVD prms;
    prms.n_cols = params.n_col2;
    prms.n_rows = params.n_row2;
    prms.n_components = params.n_col2;
    if (params.algo == 0)
      prms.algorithm = solver::COV_EIG_DQ;
    else if (params.algo == 1)
      prms.algorithm = solver::COV_EIG_JACOBI;
    else
      prms.n_components = params.n_col2 - 15;

    raft::allocate(data2, len);
    r.uniform(data2, len, T(-1.0), T(1.0), stream);
    raft::allocate(data2_trans, prms.n_rows * prms.n_components);

    int len_comp = params.n_col2 * prms.n_components;
    raft::allocate(components2, len_comp);
    raft::allocate(explained_vars2, prms.n_components);
    raft::allocate(explained_var_ratio2, prms.n_components);
    raft::allocate(singular_vals2, prms.n_components);

    tsvdFitTransform(handle, data2, data2_trans, components2, explained_vars2,
                     explained_var_ratio2, singular_vals2, prms, stream);

    raft::allocate(data2_back, len);
    tsvdInverseTransform(handle, data2_trans, components2, data2_back, prms,
                         stream);
  }

  void SetUp() override {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);
    basicTest();
    advancedTest();
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(components));
    CUDA_CHECK(hipFree(singular_vals));
    CUDA_CHECK(hipFree(components_ref));
    CUDA_CHECK(hipFree(data2));
    CUDA_CHECK(hipFree(data2_trans));
    CUDA_CHECK(hipFree(data2_back));
    CUDA_CHECK(hipFree(components2));
    CUDA_CHECK(hipFree(explained_vars2));
    CUDA_CHECK(hipFree(explained_var_ratio2));
    CUDA_CHECK(hipFree(singular_vals2));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

 protected:
  TsvdInputs<T> params;
  T *data, *components, *singular_vals, *components_ref, *explained_vars_ref;
  T *data2, *data2_trans, *data2_back, *components2, *explained_vars2,
    *explained_var_ratio2, *singular_vals2;
  raft::handle_t handle;
  hipStream_t stream;
};

const std::vector<TsvdInputs<float>> inputsf2 = {
  {0.01f, 4 * 3, 4, 3, 1024 * 128, 1024, 128, 1234ULL, 0},
  {0.01f, 4 * 3, 4, 3, 1024 * 128, 1024, 128, 1234ULL, 1},
  {0.05f, 4 * 3, 4, 3, 512 * 64, 512, 64, 1234ULL, 2},
  {0.05f, 4 * 3, 4, 3, 512 * 64, 512, 64, 1234ULL, 2}};

const std::vector<TsvdInputs<double>> inputsd2 = {
  {0.01, 4 * 3, 4, 3, 1024 * 128, 1024, 128, 1234ULL, 0},
  {0.01, 4 * 3, 4, 3, 1024 * 128, 1024, 128, 1234ULL, 1},
  {0.05, 4 * 3, 4, 3, 512 * 64, 512, 64, 1234ULL, 2},
  {0.05, 4 * 3, 4, 3, 512 * 64, 512, 64, 1234ULL, 2}};

typedef TsvdTest<float> TsvdTestLeftVecF;
TEST_P(TsvdTestLeftVecF, Result) {
  ASSERT_TRUE(
    raft::devArrMatch(components, components_ref, (params.n_col * params.n_col),
                      raft::CompareApproxAbs<float>(params.tolerance)));
}

typedef TsvdTest<double> TsvdTestLeftVecD;
TEST_P(TsvdTestLeftVecD, Result) {
  ASSERT_TRUE(
    raft::devArrMatch(components, components_ref, (params.n_col * params.n_col),
                      raft::CompareApproxAbs<double>(params.tolerance)));
}

typedef TsvdTest<float> TsvdTestDataVecF;
TEST_P(TsvdTestDataVecF, Result) {
  ASSERT_TRUE(
    raft::devArrMatch(data2, data2_back, (params.n_col2 * params.n_col2),
                      raft::CompareApproxAbs<float>(params.tolerance)));
}

typedef TsvdTest<double> TsvdTestDataVecD;
TEST_P(TsvdTestDataVecD, Result) {
  ASSERT_TRUE(
    raft::devArrMatch(data2, data2_back, (params.n_col2 * params.n_col2),
                      raft::CompareApproxAbs<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestLeftVecF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestLeftVecD,
                        ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestDataVecF,
                        ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(TsvdTests, TsvdTestDataVecD,
                        ::testing::ValuesIn(inputsd2));

}  // end namespace ML
