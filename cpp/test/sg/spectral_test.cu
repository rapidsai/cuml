/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/cluster/spectral.hpp>
#include <cuml/cuml.hpp>

#include "random/rng.h"

#include <common/cudart_utils.h>
#include <cuda_utils.h>
#include <gtest/gtest.h>
#include <test_utils.h>
#include <iostream>
#include <vector>

namespace ML {

using namespace MLCommon;

template <typename T>
class SpectralTest : public ::testing::Test {
 protected:
  void SetUp() override {}

  void TearDown() override {}

 protected:
};

typedef SpectralTest<float> TestSpectralClustering;
TEST_F(TestSpectralClustering, Fit) {
  int n = 500;
  int d = 30;
  int k = 3;

  float *X;
  cumlHandle handle;
  MLCommon::allocate(X, n * d);

  Random::Rng r(150, MLCommon::Random::GenTaps);
  r.uniform(X, n * d, -1.0f, 1.0f, handle.getStream());

  int *out;
  MLCommon::allocate(out, n, true);

  ML::Spectral::fit_clusters(handle, X, n, d, k, 10, 1e-3f, out);
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
  CUDA_CHECK(hipFree(out));
  CUDA_CHECK(hipFree(X));
}

typedef SpectralTest<float> TestSpectralEmbedding;
TEST_F(TestSpectralEmbedding, Fit) {
  int n = 500;
  int d = 30;
  int k = 3;

  float *X;
  cumlHandle handle;
  MLCommon::allocate(X, n * d);

  Random::Rng r(150, MLCommon::Random::GenTaps);
  r.uniform(X, n * d, -1.0f, 1.0f, handle.getStream());

  float *out;
  MLCommon::allocate(out, n * 2, true);

  ML::Spectral::fit_embedding(handle, X, n, d, k, 2, out);
  CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
  CUDA_CHECK(hipFree(out));
  CUDA_CHECK(hipFree(X));
}

}  // end namespace ML
