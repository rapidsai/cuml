/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/cudart_utils.h>

#include <gtest/gtest.h>
#include <raft/linalg/transpose.h>
#include <test_utils.h>
#include <cuml/datasets/make_blobs.hpp>
#include <cuml/datasets/make_regression.hpp>
#include <cuml/ensemble/randomforest.hpp>
#include <metrics/scores.cuh>

namespace ML {

using namespace MLCommon;

struct RfInputs {
  int n_rows;
  int n_cols;
  int n_trees;
  float max_features;
  float max_samples;
  int max_depth;
  int max_leaves;
  bool bootstrap;
  bool bootstrap_features;
  int n_bins;
  int split_algo;
  int min_samples_leaf;
  int min_samples_split;
  float min_impurity_decrease;
  int n_streams;
  CRITERION split_criterion;
  float min_expected_acc;
};

template <typename T>
class RFBatchedRegTest : public ::testing::TestWithParam<RfInputs> {
 protected:
  void basicTest() {
    params = ::testing::TestWithParam<RfInputs>::GetParam();

    RF_params rf_params;
    rf_params = set_rf_params(
      params.max_depth, params.max_leaves, params.max_features, params.n_bins,
      params.split_algo, params.min_samples_leaf, params.min_samples_split,
      params.min_impurity_decrease, params.bootstrap_features, params.bootstrap,
      params.n_trees, params.max_samples, 0, params.split_criterion, false,
      params.n_streams, true, 128);

    CUDA_CHECK(hipStreamCreate(&stream));
    handle.reset(new raft::handle_t(rf_params.n_streams));
    handle->set_stream(stream);
    auto allocator = handle->get_device_allocator();

    int data_len = params.n_rows * params.n_cols;
    data = (T *)allocator->allocate(data_len * sizeof(T), stream);
    data_row_major = (T *)allocator->allocate(data_len * sizeof(T), stream);
    labels = (T *)allocator->allocate(params.n_rows * sizeof(T), stream);
    predicted_labels =
      (T *)allocator->allocate(params.n_rows * sizeof(T), stream);

    Datasets::make_regression(*handle, data_row_major, labels, params.n_rows,
                              params.n_cols, params.n_cols, nullptr, 1, 0.0f,
                              -1, 0.0, 0.0f, false, 3536699ULL);

    hipblasHandle_t hipblas.h = handle->get_cublas_handle();
    raft::linalg::transpose(*handle, data_row_major, data, params.n_cols,
                            params.n_rows, stream);

    // Training part
    forest = new typename ML::RandomForestMetaData<T, T>;
    null_trees_ptr(forest);
    fit(*handle, forest, data, params.n_rows, params.n_cols, labels, rf_params);

    // predict function expects row major lay out of data, so we need to
    // transpose the data first
    predict(*handle, forest, data_row_major, params.n_rows, params.n_cols,
            predicted_labels);
    accuracy = Score::r2_score(predicted_labels, labels, params.n_rows, stream);
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    auto allocator = handle->get_device_allocator();
    allocator->deallocate(data, params.n_rows * params.n_cols * sizeof(T),
                          stream);
    allocator->deallocate(data_row_major,
                          params.n_rows * params.n_cols * sizeof(T), stream);
    allocator->deallocate(labels, params.n_rows * sizeof(T), stream);
    allocator->deallocate(predicted_labels, params.n_rows * sizeof(T), stream);
    delete forest;
    handle.reset();
  }

 protected:
  std::shared_ptr<raft::handle_t> handle;
  hipStream_t stream;
  RfInputs params;
  RandomForestMetaData<T, T> *forest;
  float accuracy = -1.0f;  // overriden in each test SetUp and TearDown
  T *data, *data_row_major;
  T *labels, *predicted_labels;
};

//-------------------------------------------------------------------------------------------------------------------------------------
const std::vector<RfInputs> inputs = {
  // Small datasets to repro corner cases as in #3107 (test for crash)
  {100, 29, 1, 1.0f, 1.0f, 2, -1, false, false, 16, SPLIT_ALGO::GLOBAL_QUANTILE,
   2, 2, 0.0, 2, CRITERION::MAE, -10.0},
  {100, 57, 2, 1.0f, 1.0f, 2, -1, false, false, 16, SPLIT_ALGO::GLOBAL_QUANTILE,
   2, 2, 0.0, 2, CRITERION::MAE, -10.0},
  {101, 57, 2, 1.0f, 1.0f, 2, -1, false, false, 13, SPLIT_ALGO::GLOBAL_QUANTILE,
   2, 2, 0.0, 2, CRITERION::MSE, -10.0},
  {100, 1, 2, 1.0f, 1.0f, 2, -1, false, false, 13, SPLIT_ALGO::GLOBAL_QUANTILE,
   2, 2, 0.0, 2, CRITERION::MAE, -10.0},

  // Larger datasets for accuracy
  {1000, 10, 10, 1.0f, 1.0f, 12, -1, true, false, 10,
   SPLIT_ALGO::GLOBAL_QUANTILE, 2, 2, 0.0, 2, CRITERION::MAE, 0.7f},
  {2000, 20, 20, 1.0f, 0.6f, 13, -1, true, false, 10,
   SPLIT_ALGO::GLOBAL_QUANTILE, 2, 2, 0.0, 2, CRITERION::MSE, 0.68f}};

typedef RFBatchedRegTest<float> RFBatchedRegTestF;
TEST_P(RFBatchedRegTestF, Fit) { ASSERT_GT(accuracy, params.min_expected_acc); }

INSTANTIATE_TEST_CASE_P(RFBatchedRegTests, RFBatchedRegTestF,
                        ::testing::ValuesIn(inputs));

typedef RFBatchedRegTest<double> RFBatchedRegTestD;
TEST_P(RFBatchedRegTestD, Fit) { ASSERT_GT(accuracy, params.min_expected_acc); }

INSTANTIATE_TEST_CASE_P(RFBatchedRegTests, RFBatchedRegTestD,
                        ::testing::ValuesIn(inputs));

}  // end namespace ML
