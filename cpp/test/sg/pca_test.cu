/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/decomposition/params.hpp>
#include <gtest/gtest.h>
#include <pca/pca.cuh>
#include <raft/cuda_utils.cuh>
#include <raft/cudart_utils.h>
#include <raft/random/rng.hpp>
#include <test_utils.h>
#include <vector>

namespace ML {

template <typename T>
struct PcaInputs {
  T tolerance;
  int len;
  int n_row;
  int n_col;
  int len2;
  int n_row2;
  int n_col2;
  unsigned long long int seed;
  int algo;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const PcaInputs<T>& dims)
{
  return os;
}

template <typename T>
class PcaTest : public ::testing::TestWithParam<PcaInputs<T>> {
 public:
  PcaTest()
    : params(::testing::TestWithParam<PcaInputs<T>>::GetParam()),
      stream(handle.get_stream()),
      explained_vars(params.n_col, stream),
      explained_vars_ref(params.n_col, stream),
      components(params.n_col * params.n_col, stream),
      components_ref(params.n_col * params.n_col, stream),
      trans_data(params.len, stream),
      trans_data_ref(params.len, stream),
      data(params.len, stream),
      data_back(params.len, stream),
      data2(params.len2, stream),
      data2_back(params.len2, stream)
  {
    basicTest();
    advancedTest();
  }

 protected:
  void basicTest()
  {
    raft::random::Rng r(params.seed, raft::random::GenPC);
    int len = params.len;

    std::vector<T> data_h = {1.0, 2.0, 5.0, 4.0, 2.0, 1.0};
    data_h.resize(len);
    raft::update_device(data.data(), data_h.data(), len, stream);

    std::vector<T> trans_data_ref_h = {-2.3231, -0.3517, 2.6748, -0.3979, 0.6571, -0.2592};
    trans_data_ref_h.resize(len);
    raft::update_device(trans_data_ref.data(), trans_data_ref_h.data(), len, stream);

    int len_comp = params.n_col * params.n_col;
    rmm::device_uvector<T> explained_var_ratio(params.n_col, stream);
    rmm::device_uvector<T> singular_vals(params.n_col, stream);
    rmm::device_uvector<T> mean(params.n_col, stream);
    rmm::device_uvector<T> noise_vars(1, stream);

    std::vector<T> components_ref_h = {0.8163, 0.5776, -0.5776, 0.8163};
    components_ref_h.resize(len_comp);
    std::vector<T> explained_vars_ref_h = {6.338, 0.3287};
    explained_vars_ref_h.resize(params.n_col);

    raft::update_device(components_ref.data(), components_ref_h.data(), len_comp, stream);
    raft::update_device(
      explained_vars_ref.data(), explained_vars_ref_h.data(), params.n_col, stream);

    paramsPCA prms;
    prms.n_cols       = params.n_col;
    prms.n_rows       = params.n_row;
    prms.n_components = params.n_col;
    prms.whiten       = false;
    if (params.algo == 0)
      prms.algorithm = solver::COV_EIG_DQ;
    else
      prms.algorithm = solver::COV_EIG_JACOBI;

    pcaFit(handle,
           data.data(),
           components.data(),
           explained_vars.data(),
           explained_var_ratio.data(),
           singular_vals.data(),
           mean.data(),
           noise_vars.data(),
           prms,
           stream);
    pcaTransform(handle,
                 data.data(),
                 components.data(),
                 trans_data.data(),
                 singular_vals.data(),
                 mean.data(),
                 prms,
                 stream);
    pcaInverseTransform(handle,
                        trans_data.data(),
                        components.data(),
                        singular_vals.data(),
                        mean.data(),
                        data_back.data(),
                        prms,
                        stream);
  }

  void advancedTest()
  {
    raft::random::Rng r(params.seed, raft::random::GenPC);
    int len = params.len2;

    paramsPCA prms;
    prms.n_cols       = params.n_col2;
    prms.n_rows       = params.n_row2;
    prms.n_components = params.n_col2;
    prms.whiten       = false;
    if (params.algo == 0)
      prms.algorithm = solver::COV_EIG_DQ;
    else if (params.algo == 1)
      prms.algorithm = solver::COV_EIG_JACOBI;

    r.uniform(data2.data(), len, T(-1.0), T(1.0), stream);
    rmm::device_uvector<T> data2_trans(prms.n_rows * prms.n_components, stream);

    int len_comp = params.n_col2 * prms.n_components;
    rmm::device_uvector<T> components2(len_comp, stream);
    rmm::device_uvector<T> explained_vars2(prms.n_components, stream);
    rmm::device_uvector<T> explained_var_ratio2(prms.n_components, stream);
    rmm::device_uvector<T> singular_vals2(prms.n_components, stream);
    rmm::device_uvector<T> mean2(prms.n_cols, stream);
    rmm::device_uvector<T> noise_vars2(1, stream);

    pcaFitTransform(handle,
                    data2.data(),
                    data2_trans.data(),
                    components2.data(),
                    explained_vars2.data(),
                    explained_var_ratio2.data(),
                    singular_vals2.data(),
                    mean2.data(),
                    noise_vars2.data(),
                    prms,
                    stream);

    pcaInverseTransform(handle,
                        data2_trans.data(),
                        components2.data(),
                        singular_vals2.data(),
                        mean2.data(),
                        data2_back.data(),
                        prms,
                        stream);
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream = 0;

  PcaInputs<T> params;

  rmm::device_uvector<T> explained_vars, explained_vars_ref, components, components_ref, trans_data,
    trans_data_ref, data, data_back, data2, data2_back;
};

const std::vector<PcaInputs<float>> inputsf2 = {
  {0.01f, 3 * 2, 3, 2, 1024 * 128, 1024, 128, 1234ULL, 0},
  {0.01f, 3 * 2, 3, 2, 256 * 32, 256, 32, 1234ULL, 1}};

const std::vector<PcaInputs<double>> inputsd2 = {
  {0.01, 3 * 2, 3, 2, 1024 * 128, 1024, 128, 1234ULL, 0},
  {0.01, 3 * 2, 3, 2, 256 * 32, 256, 32, 1234ULL, 1}};

typedef PcaTest<float> PcaTestValF;
TEST_P(PcaTestValF, Result)
{
  ASSERT_TRUE(devArrMatch(explained_vars.data(),
                          explained_vars_ref.data(),
                          params.n_col,
                          raft::CompareApproxAbs<float>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<double> PcaTestValD;
TEST_P(PcaTestValD, Result)
{
  ASSERT_TRUE(devArrMatch(explained_vars.data(),
                          explained_vars_ref.data(),
                          params.n_col,
                          raft::CompareApproxAbs<double>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<float> PcaTestLeftVecF;
TEST_P(PcaTestLeftVecF, Result)
{
  ASSERT_TRUE(devArrMatch(components.data(),
                          components_ref.data(),
                          (params.n_col * params.n_col),
                          raft::CompareApproxAbs<float>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<double> PcaTestLeftVecD;
TEST_P(PcaTestLeftVecD, Result)
{
  ASSERT_TRUE(devArrMatch(components.data(),
                          components_ref.data(),
                          (params.n_col * params.n_col),
                          raft::CompareApproxAbs<double>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<float> PcaTestTransDataF;
TEST_P(PcaTestTransDataF, Result)
{
  ASSERT_TRUE(devArrMatch(trans_data.data(),
                          trans_data_ref.data(),
                          (params.n_row * params.n_col),
                          raft::CompareApproxAbs<float>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<double> PcaTestTransDataD;
TEST_P(PcaTestTransDataD, Result)
{
  ASSERT_TRUE(devArrMatch(trans_data.data(),
                          trans_data_ref.data(),
                          (params.n_row * params.n_col),
                          raft::CompareApproxAbs<double>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<float> PcaTestDataVecSmallF;
TEST_P(PcaTestDataVecSmallF, Result)
{
  ASSERT_TRUE(devArrMatch(data.data(),
                          data_back.data(),
                          (params.n_col * params.n_col),
                          raft::CompareApproxAbs<float>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<double> PcaTestDataVecSmallD;
TEST_P(PcaTestDataVecSmallD, Result)
{
  ASSERT_TRUE(devArrMatch(data.data(),
                          data_back.data(),
                          (params.n_col * params.n_col),
                          raft::CompareApproxAbs<double>(params.tolerance),
                          handle.get_stream()));
}

// FIXME: These tests are disabled due to driver 418+ making them fail:
// https://github.com/rapidsai/cuml/issues/379
typedef PcaTest<float> PcaTestDataVecF;
TEST_P(PcaTestDataVecF, Result)
{
  ASSERT_TRUE(devArrMatch(data2.data(),
                          data2_back.data(),
                          (params.n_col2 * params.n_col2),
                          raft::CompareApproxAbs<float>(params.tolerance),
                          handle.get_stream()));
}

typedef PcaTest<double> PcaTestDataVecD;
TEST_P(PcaTestDataVecD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(data2.data(),
                                data2_back.data(),
                                (params.n_col2 * params.n_col2),
                                raft::CompareApproxAbs<double>(params.tolerance),
                                handle.get_stream()));
}

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestValF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestValD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestLeftVecF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestLeftVecD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestDataVecSmallF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestDataVecSmallD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestTransDataF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestTransDataD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestDataVecF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(PcaTests, PcaTestDataVecD, ::testing::ValuesIn(inputsd2));

}  // end namespace ML
