#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include<icecream.hpp>

#include <test_utils.h>

#include <decisiontree/quantile/quantile.h>
#include <decisiontree/batched-levelalgo/kernels.cuh>

#include <cuml/tree/algo_helper.h>
#include <cuml/datasets/make_blobs.hpp>
#include <cuml/ensemble/randomforest.hpp>

#include <random/make_blobs.cuh>

#include <raft/cudart_utils.h>
#include <raft/linalg/transpose.h>
#include <raft/cuda_utils.cuh>
#include <raft/handle.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <gtest/gtest.h>

#include <cstddef>
#include <memory>
#include <random>
#include <tuple>

namespace ML {

// Utils for changing tuple into struct
namespace detail {
template <typename result_type, typename... types, std::size_t... indices>
result_type make_struct(std::tuple<types...> t,
                        std::index_sequence<indices...>)  // &, &&, const && etc.
{
  return {std::get<indices>(t)...};
}

}  // namespace detail

template <typename result_type, typename... types>
result_type make_struct(std::tuple<types...> t)  // &, &&, const && etc.
{
  return detail::make_struct<result_type, types...>(
    t, std::index_sequence_for<types...>{});  // if there is repeated types, then the change for
                                              // using std::index_sequence_for is trivial
}

template <int I, typename RandomGenT, typename ParamT, typename T>
void SampleWithoutReplacemment(RandomGenT& gen, std::vector<ParamT>& sample, std::vector<T> x)
{
  std::vector<T> parameter_sample(sample.size());
  std::shuffle(x.begin(), x.end(), gen);
  for (size_t i = 0; i < sample.size(); i++) {
    parameter_sample[i] = x[i % x.size()];
  }
  std::shuffle(parameter_sample.begin(), parameter_sample.end(), gen);
  for (size_t i = 0; i < sample.size(); i++) {
    std::get<I>(sample[i]) = parameter_sample[i];
  }
}

template <int I, typename RandomGenT, typename ParamT, typename T, typename... Args>
void AddParameters(RandomGenT& gen, std::vector<ParamT>& sample, std::vector<T> x, Args... args)
{
  SampleWithoutReplacemment<I>(gen, sample, x);
  if constexpr (sizeof...(args) > 0) { AddParameters<I + 1>(gen, sample, args...); }
}

template <typename ParamT, typename... Args>
std::vector<ParamT> SampleParameters(int num_samples, size_t seed, Args... args)
{
  std::vector<typename ParamT::types> tuple_sample(num_samples);
  std::default_random_engine gen(seed);
  AddParameters<0>(gen, tuple_sample, args...);
  std::vector<ParamT> sample(num_samples);
  for (int i = 0; i < num_samples; i++) {
    sample[i] = make_struct<ParamT>(tuple_sample[i]);
  }
  return sample;
}

struct RfTestParams {
  int n_rows;
  int n_cols;
  int n_trees;
  float max_features;
  float max_samples;
  int max_depth;
  int max_leaves;
  bool bootstrap;
  int n_bins;
  int min_samples_leaf;
  int min_samples_split;
  float min_impurity_decrease;
  int n_streams;
  CRITERION split_criterion;
  int seed;
  int n_labels;
  bool double_precision;
  // c++ has no reflection, so we enumerate the types here
  // This must be updated if new fields are added
  using types = std::tuple<int,
                           int,
                           int,
                           float,
                           float,
                           int,
                           int,
                           bool,
                           int,
                           int,
                           int,
                           float,
                           int,
                           CRITERION,
                           int,
                           int,
                           bool>;
};

std::ostream& operator<<(std::ostream& os, const RfTestParams& ps)
{
  os << "n_rows = " << ps.n_rows << ", n_cols = " << ps.n_cols;
  os << ", n_trees = " << ps.n_trees << ", max_features = " << ps.max_features;
  os << ", max_samples = " << ps.max_samples << ", max_depth = " << ps.max_depth;
  os << ", max_leaves = " << ps.max_leaves << ", bootstrap = " << ps.bootstrap;
  os << ", n_bins = " << ps.n_bins << ", min_samples_leaf = " << ps.min_samples_leaf;
  os << ", min_samples_split = " << ps.min_samples_split;
  os << ", min_impurity_decrease = " << ps.min_impurity_decrease
     << ", n_streams = " << ps.n_streams;
  os << ", split_criterion = " << ps.split_criterion << ", seed = " << ps.seed;
  os << ", n_labels = " << ps.n_labels << ", double_precision = " << ps.double_precision;
  return os;
}

// Classification
template <typename DataT, typename LabelT, typename RfT>
void TrainScore(const raft::handle_t& handle,
                RfTestParams params,
                DataT* X,
                DataT* X_transpose,
                LabelT* y,
                RfT* forest,
                RF_metrics& metrics)
{
  RF_params rf_params = set_rf_params(params.max_depth,
                                      params.max_leaves,
                                      params.max_features,
                                      params.n_bins,
                                      params.min_samples_leaf,
                                      params.min_samples_split,
                                      params.min_impurity_decrease,
                                      params.bootstrap,
                                      params.n_trees,
                                      params.max_samples,
                                      0,
                                      params.split_criterion,
                                      params.n_streams,
                                      128);
  if constexpr (std::is_integral_v<LabelT>) {
    fit(handle, forest, X, params.n_rows, params.n_cols, y, params.n_labels, rf_params);
  } else {
    fit(handle, forest, X, params.n_rows, params.n_cols, y, rf_params);
  }

  thrust::device_vector<LabelT> pred(params.n_rows);
  predict(handle, forest, X_transpose, params.n_rows, params.n_cols, pred.data().get());
  // Predict and compare against known labels
  metrics = score(handle, forest, y, params.n_rows, pred.data().get());
}

template <typename DataT, typename LabelT>
class RfSpecialisedTest {
 public:
  RfSpecialisedTest(RfTestParams params) : params(params)
  {
    raft::handle_t handle(params.n_streams);
    X.resize(params.n_rows * params.n_cols);
    X_transpose.resize(params.n_rows * params.n_cols);
    y.resize(params.n_rows);
    // Make data
    if constexpr (std::is_integral<LabelT>::value) {
      Datasets::make_blobs(handle,
                           X.data().get(),
                           y.data().get(),
                           params.n_rows,
                           params.n_cols,
                           params.n_labels,
                           false,
                           nullptr,
                           nullptr,
                           5.0,
                           false,
                           -10.0f,
                           10.0f,
                           params.seed);
    } else {
      thrust::device_vector<int> y_temp(params.n_rows);
      Datasets::make_blobs(handle,
                           X.data().get(),
                           y_temp.data().get(),
                           params.n_rows,
                           params.n_cols,
                           params.n_labels,
                           false,
                           nullptr,
                           nullptr,
                           5.0,
                           false,
                           -10.0f,
                           10.0f,
                           params.seed);
      // if regression, make the labels normally distributed
      raft::random::Rng r(4);
      thrust::device_vector<double> normal(params.n_rows);
      r.normal(normal.data().get(), normal.size(), 0.0, 2.0, nullptr);
      thrust::transform(
        normal.begin(), normal.end(), y_temp.begin(), y.begin(), thrust::plus<LabelT>());
    }
    raft::linalg::transpose(
      handle, X.data().get(), X_transpose.data().get(), params.n_rows, params.n_cols, nullptr);
    forest.reset(new typename ML::RandomForestMetaData<DataT, LabelT>);
    TrainScore(handle,
               params,
               X.data().get(),
               X_transpose.data().get(),
               y.data().get(),
               forest.get(),
               training_metrics);

    Test();
  }
  // Current model should be at least as accurate as a model with depth - 1
  void TestAccuracyImprovement()
  {
    if (params.max_depth <= 1) { return; }
    // avereraging between models can introduce variance
    if (params.n_trees > 1) { return; }
    // accuracy is not guaranteed to improve with bootstrapping
    if (params.bootstrap) { return; }
    raft::handle_t handle(params.n_streams);
    RfTestParams alt_params = params;
    alt_params.max_depth--;
    RF_metrics alt_metrics;
    ML::RandomForestMetaData<DataT, LabelT> alt_forest;
    TrainScore(handle,
               alt_params,
               X.data().get(),
               X_transpose.data().get(),
               y.data().get(),
               &alt_forest,
               alt_metrics);
    double eps = 1e-8;
    if (params.split_criterion == MSE) {
      EXPECT_LE(training_metrics.mean_squared_error, alt_metrics.mean_squared_error + eps);
    } else if (params.split_criterion == MAE) {
      EXPECT_LE(training_metrics.mean_abs_error, alt_metrics.mean_abs_error + eps);
    } else {
      EXPECT_GE(training_metrics.accuracy, alt_metrics.accuracy);
    }
  }
  // Regularisation parameters are working correctly
  void TestTreeSize()
  {
    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      // Check we have actually built something, otherwise these tests can all pass when the tree
      // algorithm produces only stumps
      size_t effective_rows = params.n_rows * params.max_samples;
      if (params.max_depth > 0 && params.min_impurity_decrease == 0 && effective_rows >= 100) {
        EXPECT_GT(forest->trees[i].leaf_counter, 1);
      }

      // Check number of leaves is accurate
      int num_leaves = 0;
      for (auto n : forest->trees[i].sparsetree) {
        num_leaves += n.IsLeaf();
      }
      EXPECT_EQ(num_leaves, forest->trees[i].leaf_counter);
      if (params.max_leaves > 0) { EXPECT_LE(forest->trees[i].leaf_counter, params.max_leaves); }

      EXPECT_LE(forest->trees[i].depth_counter, params.max_depth);
      EXPECT_LE(forest->trees[i].leaf_counter,
                raft::ceildiv(params.n_rows, params.min_samples_leaf));
    }
  }

  void TestMinImpurity()
  {
    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      for (auto n : forest->trees[i].sparsetree) {
        if (!n.IsLeaf()) { EXPECT_GT(n.best_metric_val, params.min_impurity_decrease); }
      }
    }
  }

  void TestDeterminism()
  {
    // Regression models use floating point atomics, so are not bitwise reproducible
    bool is_regression = params.split_criterion == MSE || params.split_criterion == MAE;
    if (is_regression) return;

    // Repeat training
    RF_metrics metrics;
    raft::handle_t handle(params.n_streams);
    ML::RandomForestMetaData<DataT, LabelT> alt_forest;
    TrainScore(handle,
               params,
               X.data().get(),
               X_transpose.data().get(),
               y.data().get(),
               &alt_forest,
               metrics);

    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      EXPECT_EQ(forest->trees[i].sparsetree, alt_forest.trees[i].sparsetree);
    }
  }
  void Test()
  {
    TestAccuracyImprovement();
    TestDeterminism();
    TestTreeSize();
    TestTreeSize();
  }

  RF_metrics training_metrics;
  thrust::device_vector<DataT> X;
  thrust::device_vector<DataT> X_transpose;
  thrust::device_vector<LabelT> y;
  RfTestParams params;
  std::shared_ptr<RandomForestMetaData<DataT, LabelT>> forest;
};

// Dispatch tests based on any template parameters
class RfTest : public ::testing::TestWithParam<RfTestParams> {
 public:
  void SetUp() override
  {
    RfTestParams params = ::testing::TestWithParam<RfTestParams>::GetParam();
    bool is_regression  = params.split_criterion == MSE || params.split_criterion == MAE ||
                         params.split_criterion == POISSON;
    if (params.double_precision) {
      if (is_regression) {
        RfSpecialisedTest<double, double> test(params);
      } else {
        RfSpecialisedTest<double, int> test(params);
      }
    } else {
      if (is_regression) {
        RfSpecialisedTest<float, float> test(params);
      } else {
        RfSpecialisedTest<float, int> test(params);
      }
    }
  }
};

TEST_P(RfTest, PropertyBasedTest) {}

// Parameter ranges to test
std::vector<int> n_rows                  = {10, 100, 1452};
std::vector<int> n_cols                  = {1, 5, 152, 1014};
std::vector<int> n_trees                 = {1, 5, 17};
std::vector<float> max_features          = {0.1f, 0.5f, 1.0f};
std::vector<float> max_samples           = {0.1f, 0.5f, 1.0f};
std::vector<int> max_depth               = {1, 10, 30};
std::vector<int> max_leaves              = {-1, 16, 50};
std::vector<bool> bootstrap              = {false, true};
std::vector<int> n_bins                  = {2, 57, 128, 256};
std::vector<int> min_samples_leaf        = {1, 10, 30};
std::vector<int> min_samples_split       = {2, 10};
std::vector<float> min_impurity_decrease = {0.0f, 1.0f, 10.0f};
std::vector<int> n_streams               = {1, 2, 10};
std::vector<CRITERION> split_criterion   = {
  CRITERION::POISSON, CRITERION::MSE, CRITERION::GINI, CRITERION::ENTROPY};
std::vector<int> seed              = {0, 17};
std::vector<int> n_labels          = {2, 10, 30};
std::vector<bool> double_precision = {false, true};

int n_tests = 100;

INSTANTIATE_TEST_CASE_P(RfTests,
                        RfTest,
                        ::testing::ValuesIn(SampleParameters<RfTestParams>(n_tests,
                                                                           0,
                                                                           n_rows,
                                                                           n_cols,
                                                                           n_trees,
                                                                           max_features,
                                                                           max_samples,
                                                                           max_depth,
                                                                           max_leaves,
                                                                           bootstrap,
                                                                           n_bins,
                                                                           min_samples_leaf,
                                                                           min_samples_split,
                                                                           min_impurity_decrease,
                                                                           n_streams,
                                                                           split_criterion,
                                                                           seed,
                                                                           n_labels,
                                                                           double_precision)));

struct QuantileTestParameters {
  int n_rows;
  int n_bins;
  uint64_t seed;
};

template <typename T>
class RFQuantileBinsLowerBoundTest : public ::testing::TestWithParam<QuantileTestParameters> {
 public:
  void SetUp() override
  {
    auto params = ::testing::TestWithParam<QuantileTestParameters>::GetParam();

    thrust::device_vector<T> data(params.n_rows);
    thrust::host_vector<T> h_data(params.n_rows);
    thrust::device_vector<T> quantiles(params.n_bins);
    thrust::host_vector<T> h_quantiles(params.n_bins);
    raft::random::Rng r(8);
    r.normal(data.data().get(), data.size(), T(0.0), T(2.0), nullptr);
    raft::handle_t handle;
    DT::computeQuantiles(
      quantiles.data().get(), params.n_bins, data.data().get(), params.n_rows, 1, nullptr);
    h_quantiles = quantiles;
    h_data      = data;
    for (std::size_t i = 0; i < h_data.size(); ++i) {
      auto d = h_data[i];
      // golden lower bound from thrust
      auto golden_lb = thrust::lower_bound(
                         thrust::seq, h_quantiles.data(), h_quantiles.data() + params.n_bins, d) -
                       h_quantiles.data();
      // lower bound from custom lower_bound impl
      auto lb = DT::lower_bound(h_quantiles.data(), params.n_bins, d);
      ASSERT_EQ(golden_lb, lb)
        << "custom lower_bound method is inconsistent with thrust::lower_bound" << std::endl;
    }
  }
};

template <typename T>
class RFQuantileTest : public ::testing::TestWithParam<QuantileTestParameters> {
 public:
  void SetUp() override
  {
    auto params = ::testing::TestWithParam<QuantileTestParameters>::GetParam();

    thrust::device_vector<T> data(params.n_rows);
    thrust::device_vector<T> quantiles(params.n_bins);
    thrust::device_vector<int> histogram(params.n_bins);
    thrust::host_vector<int> h_histogram(params.n_bins);

    raft::random::Rng r(8);
    r.normal(data.data().get(), data.size(), T(0.0), T(2.0), nullptr);
    raft::handle_t handle;
    DT::computeQuantiles(
      quantiles.data().get(), params.n_bins, data.data().get(), params.n_rows, 1, nullptr);

    auto d_quantiles = quantiles.data();
    auto d_histogram = histogram.data().get();
    thrust::for_each(data.begin(), data.end(), [=] __device__(T x) {
      for (int j = 0; j < params.n_bins; j++) {
        if (x <= d_quantiles[j]) {
          atomicAdd(&d_histogram[j], 1);
          break;
        }
      }
    });

    h_histogram           = histogram;
    int max_items_per_bin = raft::ceildiv(params.n_rows, params.n_bins);
    int min_items_per_bin = max_items_per_bin - 1;
    int total_items       = 0;
    for (int b = 0; b < params.n_bins; b++) {
      ASSERT_TRUE(h_histogram[b] == max_items_per_bin || h_histogram[b] == min_items_per_bin)
        << "No. samples in bin[" << b << "] = " << h_histogram[b] << " Expected "
        << max_items_per_bin << " or " << min_items_per_bin << std::endl;
      total_items += h_histogram[b];
    }
    ASSERT_EQ(params.n_rows, total_items)
      << "Some samples from dataset are either missed of double counted in quantile bins"
      << std::endl;
  }
};

//-------------------------------------------------------------------------------------------------------------------------------------
const std::vector<QuantileTestParameters> inputs = {{1000, 16, 6078587519764079670LLU},
                                                    {1130, 32, 4884670006177930266LLU},
                                                    {1752, 67, 9175325892580481371LLU},
                                                    {2307, 99, 9507819643927052255LLU},
                                                    {5000, 128, 9507819643927052255LLU}};

// float type quantile test
typedef RFQuantileTest<float> RFQuantileTestF;
TEST_P(RFQuantileTestF, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileTestF, ::testing::ValuesIn(inputs));

// double type quantile test
typedef RFQuantileTest<double> RFQuantileTestD;
TEST_P(RFQuantileTestD, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileTestD, ::testing::ValuesIn(inputs));

// float type quantile bins lower bounds test
typedef RFQuantileBinsLowerBoundTest<float> RFQuantileBinsLowerBoundTestF;
TEST_P(RFQuantileBinsLowerBoundTestF, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileBinsLowerBoundTestF, ::testing::ValuesIn(inputs));

// double type quantile bins lower bounds lest
typedef RFQuantileBinsLowerBoundTest<double> RFQuantileBinsLowerBoundTestD;
TEST_P(RFQuantileBinsLowerBoundTestD, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileBinsLowerBoundTestD, ::testing::ValuesIn(inputs));

//------------------------------------------------------------------------------------------------------

namespace DT {

struct ObjectiveTestParameters
{
  CRITERION criterion;
  uint64_t seed;
  int n_bins;
  int n_classes;
  double min_impurity_decrease;
  int min_samples_leaf;

};

template <typename ObjectiveT>
class ObjectiveTest : public ::testing::TestWithParam<ObjectiveTestParameters>
{
  typedef typename ObjectiveT::DataT DataT;
  typedef typename ObjectiveT::LabelT LabelT;
  typedef typename ObjectiveT::IdxT IdxT;
  typedef typename ObjectiveT::BinT BinT;

  ObjectiveTestParameters params;

 public:

 auto _rand(int const end = 1000)
 {
   return rand() % end;
 }

  auto _gen_hist_bins(){
    std::vector<BinT> hist_bins(params.n_bins * params.n_classes);
    for(auto c = 0; c < params.n_classes; ++c)
    {
      for(auto b = 0; b < params.n_bins; ++b)
      {
        // initializing hist_bins
        BinT tmp = BinT();
        if constexpr(std::is_same<BinT, CountBin>::value) // classification type
        {
          tmp += BinT(_rand());
          hist_bins[c*params.n_bins + b] += tmp; // random pdf bin
          hist_bins[c*params.n_bins + b] += ( b > 0 ? hist_bins[c*params.n_bins + b - 1] : BinT()); // pdf to cdf
        }
        else // regression type
        {
          tmp += BinT(static_cast<LabelT>(_rand()), _rand());
          hist_bins[c*params.n_bins + b] += tmp; // random pdf bin
          hist_bins[c*params.n_bins + b] += ( b > 0 ? hist_bins[c*params.n_bins + b - 1] : BinT()); // pdf to cdf
        }
      }
    }
    return hist_bins;
  }

  auto _poisson_ground_truth_gain(std::vector<BinT> const & hist_bins, std::size_t split_bin_index)
  {

    // compute the gain to be
    DataT label_sum      = hist_bins.back().label_sum;
    IdxT len            = hist_bins.back().count;
    IdxT nLeft          = hist_bins[split_bin_index].count;
    DataT left_label_sum = hist_bins[split_bin_index].label_sum;
    DataT right_label_sum= label_sum - left_label_sum;
    IdxT nRight         = len - nLeft;
    DataT parent_obj     = -label_sum * raft::myLog(label_sum / len);
    DataT left_obj       = -left_label_sum * raft::myLog(left_label_sum / nLeft);
    DataT right_obj      = -right_label_sum * raft::myLog(right_label_sum / nRight);
    auto gain      = parent_obj - (left_obj + right_obj);
    gain           = gain / len;

    // edge cases
    if (gain <= params.min_impurity_decrease ||
        nLeft < params.min_samples_leaf ||
        nRight < params.min_samples_leaf ||
        label_sum < EPS ||
        right_label_sum < EPS ||
        left_label_sum < EPS)
      return -std::numeric_limits<DataT>::max();
    else return gain;

  }

  auto _gini_ground_truth_gain(std::vector<BinT> const & hist_bins, std::size_t const split_bin_index)
  {
    auto len = _get_nLeft(hist_bins, params.n_bins-1);
    auto nLeft = _get_nLeft(hist_bins, split_bin_index);
    auto nRight         = len - nLeft;
    constexpr DataT One = DataT(1.0);
    auto invlen         = One / len;
    auto invLeft        = One / nLeft;
    auto invRight       = One / nRight;
    auto gain           = DataT(0.0);

    for(IdxT c = 0; c < params.n_classes; ++c)
    {
      IdxT val_i  = 0;
      auto lval_i = hist_bins[params.n_bins * c + split_bin_index].x;
      auto lval   = DataT(lval_i);
      gain += lval * invLeft * lval * invlen;

      val_i += lval_i;
      auto total_sum = hist_bins[params.n_bins * c + params.n_bins - 1].x;
      auto rval_i    = total_sum - lval_i;
      auto rval      = DataT(rval_i);
      gain += rval * invRight * rval * invlen;

      val_i += rval_i;
      auto val = DataT(val_i) * invlen;
      gain -= val * val;
    }

    // edge cases
    if (gain <= params.min_impurity_decrease ||
        nLeft < params.min_samples_leaf ||
        nRight < params.min_samples_leaf)
    {
      return -std::numeric_limits<DataT>::max();
    }
    else
    {
      return gain;
    }
  }

  auto _get_ground_truth_gain(std::vector<BinT> const & hist_bins, std::size_t const split_bin_index)
  {
    if constexpr(std::is_same<ObjectiveT, PoissonObjectiveFunction<DataT, LabelT, IdxT>>::value) // poisson
    {
      return _poisson_ground_truth_gain(hist_bins, split_bin_index);
    }
    else if constexpr(std::is_same<ObjectiveT, GiniObjectiveFunction<DataT, LabelT, IdxT>>::value) // gini
    {
      return _gini_ground_truth_gain(hist_bins, split_bin_index);
    }
    return (double)0.0;
  }

  auto _get_nLeft(std::vector<BinT> const & hist_bins, IdxT idx)
  {
    auto count {IdxT(0)};
    for (auto c = 0; c < params.n_classes; ++c)
    {
      if constexpr(std::is_same<BinT, CountBin>::value) // countbin
      {
        count += hist_bins[params.n_bins * c + idx].x;
      }
      else // aggregatebin
      {
        count += hist_bins[params.n_bins * c + idx].count;
      }
    }
    return count;
  }

  void SetUp() override
  {
    srand(params.seed);
    params = ::testing::TestWithParam<ObjectiveTestParameters>::GetParam();
    ObjectiveT objective(params.n_classes, params.min_impurity_decrease, params.min_samples_leaf);

    auto hist_bins = _gen_hist_bins();
    auto split_bin_index = _rand(params.n_bins);
    auto ground_truth_gain = _get_ground_truth_gain(hist_bins, split_bin_index);
    auto hypothesis_gain = objective.gain(&hist_bins[0],
                                          split_bin_index,
                                          params.n_bins,
                                          _get_nLeft(hist_bins, params.n_bins-1),
                                          _get_nLeft(hist_bins, split_bin_index));

    ASSERT_EQ(ground_truth_gain, hypothesis_gain);

  }
};

const std::vector<ObjectiveTestParameters> poisson_objective_test_parameters = {
                                                                               {CRITERION::POISSON, 9507819643927052255LLU, 64, 1, 0.0001, 0},
                                                                               {CRITERION::POISSON, 9507819643927052256LLU, 128, 1, 0.0001, 1},
                                                                               {CRITERION::POISSON, 9507819643927052257LLU, 256, 1, 0.0001, 1},
                                                                               {CRITERION::POISSON, 9507819643927052258LLU, 512, 1, 0.0001, 5},
                                                                               };
const std::vector<ObjectiveTestParameters> gini_objective_test_parameters = {
                                                                            {CRITERION::GINI, 9507819643927052255LLU, 64, 2, 0.0001, 0},
                                                                            {CRITERION::GINI, 9507819643927052256LLU, 128, 10, 0.0001, 1},
                                                                            {CRITERION::GINI, 9507819643927052257LLU, 256, 100, 0.0001, 1},
                                                                            {CRITERION::GINI, 9507819643927052258LLU, 512, 100, 0.0001, 5},
                                                                            };

// poisson objective test
typedef ObjectiveTest<PoissonObjectiveFunction<double, double, int>> PoissonObjectiveTestD;
TEST_P(PoissonObjectiveTestD, poissonObjectiveTest) {}
INSTANTIATE_TEST_CASE_P(RfTests, PoissonObjectiveTestD, ::testing::ValuesIn(poisson_objective_test_parameters));

// gini objective test
typedef ObjectiveTest<GiniObjectiveFunction<double, double, int>> GiniObjectiveTestD;
TEST_P(GiniObjectiveTestD, giniObjectiveTest) {}
INSTANTIATE_TEST_CASE_P(RfTests, GiniObjectiveTestD, ::testing::ValuesIn(gini_objective_test_parameters));

} // end namespace DT
}  // end namespace ML
