#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <test_utils.h>

#include <decisiontree/batched-levelalgo/kernels.cuh>
#include <decisiontree/batched-levelalgo/quantile.cuh>

#include <cuml/fil/fil.h>
#include <cuml/tree/algo_helper.h>
#include <cuml/datasets/make_blobs.hpp>
#include <cuml/ensemble/randomforest.hpp>

#include <random/make_blobs.cuh>

#include <raft/cudart_utils.h>
#include <raft/linalg/transpose.h>
#include <raft/cuda_utils.cuh>
#include <raft/handle.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/logical.h>

#include <gtest/gtest.h>

#include <cstddef>
#include <memory>
#include <random>
#include <tuple>
#include <type_traits>

namespace ML {

// Utils for changing tuple into struct
namespace detail {
template <typename result_type, typename... types, std::size_t... indices>
result_type make_struct(std::tuple<types...> t,
                        std::index_sequence<indices...>)  // &, &&, const && etc.
{
  return {std::get<indices>(t)...};
}

}  // namespace detail

template <typename result_type, typename... types>
result_type make_struct(std::tuple<types...> t)  // &, &&, const && etc.
{
  return detail::make_struct<result_type, types...>(
    t, std::index_sequence_for<types...>{});  // if there is repeated types, then the change for
                                              // using std::index_sequence_for is trivial
}

template <int I, typename RandomGenT, typename ParamT, typename T>
void SampleWithoutReplacemment(RandomGenT& gen, std::vector<ParamT>& sample, std::vector<T> x)
{
  std::vector<T> parameter_sample(sample.size());
  std::shuffle(x.begin(), x.end(), gen);
  for (size_t i = 0; i < sample.size(); i++) {
    parameter_sample[i] = x[i % x.size()];
  }
  std::shuffle(parameter_sample.begin(), parameter_sample.end(), gen);
  for (size_t i = 0; i < sample.size(); i++) {
    std::get<I>(sample[i]) = parameter_sample[i];
  }
}

template <int I, typename RandomGenT, typename ParamT, typename T, typename... Args>
void AddParameters(RandomGenT& gen, std::vector<ParamT>& sample, std::vector<T> x, Args... args)
{
  SampleWithoutReplacemment<I>(gen, sample, x);
  if constexpr (sizeof...(args) > 0) { AddParameters<I + 1>(gen, sample, args...); }
}

template <typename ParamT, typename... Args>
std::vector<ParamT> SampleParameters(int num_samples, size_t seed, Args... args)
{
  std::vector<typename ParamT::types> tuple_sample(num_samples);
  std::default_random_engine gen(seed);
  AddParameters<0>(gen, tuple_sample, args...);
  std::vector<ParamT> sample(num_samples);
  for (int i = 0; i < num_samples; i++) {
    sample[i] = make_struct<ParamT>(tuple_sample[i]);
  }
  return sample;
}

struct RfTestParams {
  int n_rows;
  int n_cols;
  int n_trees;
  float max_features;
  float max_samples;
  int max_depth;
  int max_leaves;
  bool bootstrap;
  int n_bins;
  int min_samples_leaf;
  int min_samples_split;
  float min_impurity_decrease;
  int n_streams;
  CRITERION split_criterion;
  int seed;
  int n_labels;
  bool double_precision;
  // c++ has no reflection, so we enumerate the types here
  // This must be updated if new fields are added
  using types = std::tuple<int,
                           int,
                           int,
                           float,
                           float,
                           int,
                           int,
                           bool,
                           int,
                           int,
                           int,
                           float,
                           int,
                           CRITERION,
                           int,
                           int,
                           bool>;
};

std::ostream& operator<<(std::ostream& os, const RfTestParams& ps)
{
  os << "n_rows = " << ps.n_rows << ", n_cols = " << ps.n_cols;
  os << ", n_trees = " << ps.n_trees << ", max_features = " << ps.max_features;
  os << ", max_samples = " << ps.max_samples << ", max_depth = " << ps.max_depth;
  os << ", max_leaves = " << ps.max_leaves << ", bootstrap = " << ps.bootstrap;
  os << ", n_bins = " << ps.n_bins << ", min_samples_leaf = " << ps.min_samples_leaf;
  os << ", min_samples_split = " << ps.min_samples_split;
  os << ", min_impurity_decrease = " << ps.min_impurity_decrease
     << ", n_streams = " << ps.n_streams;
  os << ", split_criterion = " << ps.split_criterion << ", seed = " << ps.seed;
  os << ", n_labels = " << ps.n_labels << ", double_precision = " << ps.double_precision;
  return os;
}

template <typename DataT, typename LabelT>
auto FilPredict(const raft::handle_t& handle,
                RfTestParams params,
                DataT* X_transpose,
                RandomForestMetaData<DataT, LabelT>* forest)
{
  auto pred = std::make_shared<thrust::device_vector<float>>(params.n_rows);
  ModelHandle model;
  std::size_t num_outputs = 1;
  if constexpr (std::is_integral_v<LabelT>) { num_outputs = params.n_labels; }
  build_treelite_forest(&model, forest, params.n_cols);
  fil::treelite_params_t tl_params{fil::algo_t::ALGO_AUTO,
                                   num_outputs > 1,
                                   1.f / num_outputs,
                                   fil::storage_type_t::AUTO,
                                   8,
                                   1,
                                   0,
                                   nullptr};
  fil::forest_t fil_forest;
  fil::from_treelite(handle, &fil_forest, model, &tl_params);
  fil::predict(handle, fil_forest, pred->data().get(), X_transpose, params.n_rows, false);
  return pred;
}

template <typename DataT, typename LabelT>
auto FilPredictProba(const raft::handle_t& handle,
                     RfTestParams params,
                     DataT* X_transpose,
                     RandomForestMetaData<DataT, LabelT>* forest)
{
  std::size_t num_outputs = params.n_labels;
  auto pred = std::make_shared<thrust::device_vector<float>>(params.n_rows * num_outputs);
  ModelHandle model;
  static_assert(std::is_integral_v<LabelT>, "Must be classification");
  build_treelite_forest(&model, forest, params.n_cols);
  fil::treelite_params_t tl_params{
    fil::algo_t::ALGO_AUTO, 0, 0.0f, fil::storage_type_t::AUTO, 8, 1, 0, nullptr};
  fil::forest_t fil_forest;
  fil::from_treelite(handle, &fil_forest, model, &tl_params);
  fil::predict(handle, fil_forest, pred->data().get(), X_transpose, params.n_rows, true);
  return pred;
}
template <typename DataT, typename LabelT>
auto TrainScore(
  const raft::handle_t& handle, RfTestParams params, DataT* X, DataT* X_transpose, LabelT* y)
{
  RF_params rf_params = set_rf_params(params.max_depth,
                                      params.max_leaves,
                                      params.max_features,
                                      params.n_bins,
                                      params.min_samples_leaf,
                                      params.min_samples_split,
                                      params.min_impurity_decrease,
                                      params.bootstrap,
                                      params.n_trees,
                                      params.max_samples,
                                      0,
                                      params.split_criterion,
                                      params.n_streams,
                                      128);

  auto forest     = std::make_shared<RandomForestMetaData<DataT, LabelT>>();
  auto forest_ptr = forest.get();
  if constexpr (std::is_integral_v<LabelT>) {
    fit(handle, forest_ptr, X, params.n_rows, params.n_cols, y, params.n_labels, rf_params);
  } else {
    fit(handle, forest_ptr, X, params.n_rows, params.n_cols, y, rf_params);
  }

  auto pred = std::make_shared<thrust::device_vector<LabelT>>(params.n_rows);
  predict(handle, forest_ptr, X_transpose, params.n_rows, params.n_cols, pred->data().get());

  // Predict and compare against known labels
  RF_metrics metrics = score(handle, forest_ptr, y, params.n_rows, pred->data().get());
  return std::make_tuple(forest, pred, metrics);
}

template <typename DataT, typename LabelT>
class RfSpecialisedTest {
 public:
  RfSpecialisedTest(RfTestParams params) : params(params)
  {
    raft::handle_t handle(params.n_streams);
    X.resize(params.n_rows * params.n_cols);
    X_transpose.resize(params.n_rows * params.n_cols);
    y.resize(params.n_rows);
    // Make data
    if constexpr (std::is_integral<LabelT>::value) {
      Datasets::make_blobs(handle,
                           X.data().get(),
                           y.data().get(),
                           params.n_rows,
                           params.n_cols,
                           params.n_labels,
                           false,
                           nullptr,
                           nullptr,
                           5.0,
                           false,
                           -10.0f,
                           10.0f,
                           params.seed);
    } else {
      thrust::device_vector<int> y_temp(params.n_rows);
      Datasets::make_blobs(handle,
                           X.data().get(),
                           y_temp.data().get(),
                           params.n_rows,
                           params.n_cols,
                           params.n_labels,
                           false,
                           nullptr,
                           nullptr,
                           5.0,
                           false,
                           -10.0f,
                           10.0f,
                           params.seed);
      // if regression, make the labels normally distributed
      raft::random::Rng r(4);
      thrust::device_vector<double> normal(params.n_rows);
      r.normal(normal.data().get(), normal.size(), 0.0, 2.0, nullptr);
      thrust::transform(
        normal.begin(), normal.end(), y_temp.begin(), y.begin(), thrust::plus<LabelT>());
    }
    raft::linalg::transpose(
      handle, X.data().get(), X_transpose.data().get(), params.n_rows, params.n_cols, nullptr);
    forest.reset(new typename ML::RandomForestMetaData<DataT, LabelT>);
    std::tie(forest, predictions, training_metrics) =
      TrainScore(handle, params, X.data().get(), X_transpose.data().get(), y.data().get());

    Test();
  }
  // Current model should be at least as accurate as a model with depth - 1
  void TestAccuracyImprovement()
  {
    if (params.max_depth <= 1) { return; }
    // avereraging between models can introduce variance
    if (params.n_trees > 1) { return; }
    // accuracy is not guaranteed to improve with bootstrapping
    if (params.bootstrap) { return; }
    raft::handle_t handle(params.n_streams);
    RfTestParams alt_params = params;
    alt_params.max_depth--;
    auto [alt_forest, alt_predictions, alt_metrics] =
      TrainScore(handle, alt_params, X.data().get(), X_transpose.data().get(), y.data().get());
    double eps = 1e-8;
    if (params.split_criterion == MSE) {
      EXPECT_LE(training_metrics.mean_squared_error, alt_metrics.mean_squared_error + eps);
    } else if (params.split_criterion == MAE) {
      EXPECT_LE(training_metrics.mean_abs_error, alt_metrics.mean_abs_error + eps);
    } else {
      EXPECT_GE(training_metrics.accuracy, alt_metrics.accuracy);
    }
  }
  // Regularisation parameters are working correctly
  void TestTreeSize()
  {
    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      // Check we have actually built something, otherwise these tests can all pass when the tree
      // algorithm produces only stumps
      size_t effective_rows = params.n_rows * params.max_samples;
      if (params.max_depth > 0 && params.min_impurity_decrease == 0 && effective_rows >= 100) {
        EXPECT_GT(forest->trees[i]->leaf_counter, 1);
      }

      // Check number of leaves is accurate
      int num_leaves = 0;
      for (auto n : forest->trees[i]->sparsetree) {
        num_leaves += n.IsLeaf();
      }
      EXPECT_EQ(num_leaves, forest->trees[i]->leaf_counter);
      if (params.max_leaves > 0) { EXPECT_LE(forest->trees[i]->leaf_counter, params.max_leaves); }

      EXPECT_LE(forest->trees[i]->depth_counter, params.max_depth);
      EXPECT_LE(forest->trees[i]->leaf_counter,
                raft::ceildiv(params.n_rows, params.min_samples_leaf));
    }
  }
  void TestMinImpurity()
  {
    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      for (auto n : forest->trees[i]->sparsetree) {
        if (!n.IsLeaf()) { EXPECT_GT(n.BestMetric(), params.min_impurity_decrease); }
      }
    }
  }
  void TestDeterminism()

  {
    // Regression models use floating point atomics, so are not bitwise reproducible
    bool is_regression = params.split_criterion == MSE || params.split_criterion == MAE;
    if (is_regression) return;

    // Repeat training
    raft::handle_t handle(params.n_streams);
    auto [alt_forest, alt_predictions, alt_metrics] =
      TrainScore(handle, params, X.data().get(), X_transpose.data().get(), y.data().get());

    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      EXPECT_EQ(forest->trees[i]->sparsetree, alt_forest->trees[i]->sparsetree);
    }
  }
  // Instance counts in children sums up to parent.
  void TestInstanceCounts()
  {
    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      const auto& tree = forest->trees[i]->sparsetree;
      for (auto n : tree) {
        if (!n.IsLeaf()) {
          auto sum = tree[n.LeftChildId()].InstanceCount() + tree[n.RightChildId()].InstanceCount();
          EXPECT_EQ(sum, n.InstanceCount());
        }
      }
    }
  }

  // Difference between the largest element and second largest
  DataT MinDifference(DataT* begin, std::size_t len)
  {
    std::size_t max_element_index = 0;
    DataT max_element             = 0.0;
    for (std::size_t i = 0; i < len; i++) {
      if (begin[i] > max_element) {
        max_element_index = i;
        max_element       = begin[i];
      }
    }
    DataT second_max_element = 0.0;
    for (std::size_t i = 0; i < len; i++) {
      if (begin[i] > second_max_element && i != max_element_index) {
        second_max_element = begin[i];
      }
    }

    return std::abs(max_element - second_max_element);
  }

  // Compare fil against native rf predictions
  // Only for single precision models
  void TestFilPredict()
  {
    if constexpr (std::is_same_v<DataT, double>) {
      return;
    } else {
      raft::handle_t handle(params.n_streams);
      auto fil_pred = FilPredict(handle, params, X_transpose.data().get(), forest.get());

      thrust::host_vector<float> h_fil_pred(*fil_pred);
      thrust::host_vector<float> h_pred(*predictions);

      thrust::host_vector<float> h_fil_pred_prob;
      if constexpr (std::is_integral_v<LabelT>) {
        h_fil_pred_prob = *FilPredictProba(handle, params, X_transpose.data().get(), forest.get());
      }

      float tol = 1e-2;
      for (std::size_t i = 0; i < h_fil_pred.size(); i++) {
        // If the output probabilities are very similar for different classes
        // FIL may output a different class due to numerical differences
        // Skip these cases
        if constexpr (std::is_integral_v<LabelT>) {
          int num_outputs = forest->trees[0]->num_outputs;
          auto min_diff   = MinDifference(&h_fil_pred_prob[i * num_outputs], num_outputs);
          if (min_diff < tol) continue;
        }

        EXPECT_LE(abs(h_fil_pred[i] - h_pred[i]), tol);
      }
    }
  }
  void Test()
  {
    TestAccuracyImprovement();
    TestDeterminism();
    TestMinImpurity();
    TestTreeSize();
    TestInstanceCounts();
    TestFilPredict();
  }

  RF_metrics training_metrics;
  thrust::device_vector<DataT> X;
  thrust::device_vector<DataT> X_transpose;
  thrust::device_vector<LabelT> y;
  RfTestParams params;
  std::shared_ptr<RandomForestMetaData<DataT, LabelT>> forest;
  std::shared_ptr<thrust::device_vector<LabelT>> predictions;
};

// Dispatch tests based on any template parameters
class RfTest : public ::testing::TestWithParam<RfTestParams> {
 public:
  void SetUp() override
  {
    RfTestParams params = ::testing::TestWithParam<RfTestParams>::GetParam();
    bool is_regression  = params.split_criterion == MSE || params.split_criterion == MAE;
    if (params.double_precision) {
      if (is_regression) {
        RfSpecialisedTest<double, double> test(params);
      } else {
        RfSpecialisedTest<double, int> test(params);
      }
    } else {
      if (is_regression) {
        RfSpecialisedTest<float, float> test(params);
      } else {
        RfSpecialisedTest<float, int> test(params);
      }
    }
  }
};

TEST_P(RfTest, PropertyBasedTest) {}

// Parameter ranges to test
std::vector<int> n_rows                  = {10, 100, 1452};
std::vector<int> n_cols                  = {1, 5, 152, 1014};
std::vector<int> n_trees                 = {1, 5, 17};
std::vector<float> max_features          = {0.1f, 0.5f, 1.0f};
std::vector<float> max_samples           = {0.1f, 0.5f, 1.0f};
std::vector<int> max_depth               = {1, 10, 30};
std::vector<int> max_leaves              = {-1, 16, 50};
std::vector<bool> bootstrap              = {false, true};
std::vector<int> n_bins                  = {2, 57, 128, 256};
std::vector<int> min_samples_leaf        = {1, 10, 30};
std::vector<int> min_samples_split       = {2, 10};
std::vector<float> min_impurity_decrease = {0.0f, 1.0f, 10.0f};
std::vector<int> n_streams               = {1, 2, 10};
std::vector<CRITERION> split_criterion   = {CRITERION::MSE, CRITERION::GINI, CRITERION::ENTROPY};
std::vector<int> seed                    = {0, 17};
std::vector<int> n_labels                = {2, 10, 20};
std::vector<bool> double_precision       = {false, true};

int n_tests = 100;

INSTANTIATE_TEST_CASE_P(RfTests,
                        RfTest,
                        ::testing::ValuesIn(SampleParameters<RfTestParams>(n_tests,
                                                                           0,
                                                                           n_rows,
                                                                           n_cols,
                                                                           n_trees,
                                                                           max_features,
                                                                           max_samples,
                                                                           max_depth,
                                                                           max_leaves,
                                                                           bootstrap,
                                                                           n_bins,
                                                                           min_samples_leaf,
                                                                           min_samples_split,
                                                                           min_impurity_decrease,
                                                                           n_streams,
                                                                           split_criterion,
                                                                           seed,
                                                                           n_labels,
                                                                           double_precision)));

struct QuantileTestParameters {
  int n_rows;
  int n_bins;
  uint64_t seed;
};

template <typename T>
class RFQuantileBinsLowerBoundTest : public ::testing::TestWithParam<QuantileTestParameters> {
 public:
  void SetUp() override
  {
    auto params = ::testing::TestWithParam<QuantileTestParameters>::GetParam();

    thrust::device_vector<T> data(params.n_rows);
    thrust::host_vector<T> h_data(params.n_rows);
    thrust::host_vector<T> h_quantiles(params.n_bins);
    raft::random::Rng r(8);
    r.normal(data.data().get(), data.size(), T(0.0), T(2.0), nullptr);
    raft::handle_t handle;
    auto quantiles =
      DT::computeQuantiles(params.n_bins, data.data().get(), params.n_rows, 1, handle);
    raft::update_host(
      h_quantiles.data(), quantiles->data(), quantiles->size(), handle.get_stream());
    h_data = data;
    for (std::size_t i = 0; i < h_data.size(); ++i) {
      auto d = h_data[i];
      // golden lower bound from thrust
      auto golden_lb = thrust::lower_bound(
                         thrust::seq, h_quantiles.data(), h_quantiles.data() + params.n_bins, d) -
                       h_quantiles.data();
      // lower bound from custom lower_bound impl
      auto lb = DT::lower_bound(h_quantiles.data(), params.n_bins, d);
      ASSERT_EQ(golden_lb, lb)
        << "custom lower_bound method is inconsistent with thrust::lower_bound" << std::endl;
    }
  }
};

template <typename T>
class RFQuantileTest : public ::testing::TestWithParam<QuantileTestParameters> {
 public:
  void SetUp() override
  {
    auto params = ::testing::TestWithParam<QuantileTestParameters>::GetParam();

    thrust::device_vector<T> data(params.n_rows);
    thrust::device_vector<int> histogram(params.n_bins);
    thrust::host_vector<int> h_histogram(params.n_bins);

    raft::random::Rng r(8);
    r.normal(data.data().get(), data.size(), T(0.0), T(2.0), nullptr);
    raft::handle_t handle;
    std::shared_ptr<rmm::device_uvector<T>> quantiles =
      DT::computeQuantiles(params.n_bins, data.data().get(), params.n_rows, 1, handle);

    auto d_quantiles = quantiles->data();
    auto d_histogram = histogram.data().get();
    thrust::for_each(data.begin(), data.end(), [=] __device__(T x) {
      for (int j = 0; j < params.n_bins; j++) {
        if (x <= d_quantiles[j]) {
          atomicAdd(&d_histogram[j], 1);
          break;
        }
      }
    });

    h_histogram           = histogram;
    int max_items_per_bin = raft::ceildiv(params.n_rows, params.n_bins);
    int min_items_per_bin = max_items_per_bin - 1;
    int total_items       = 0;
    for (int b = 0; b < params.n_bins; b++) {
      ASSERT_TRUE(h_histogram[b] == max_items_per_bin || h_histogram[b] == min_items_per_bin)
        << "No. samples in bin[" << b << "] = " << h_histogram[b] << " Expected "
        << max_items_per_bin << " or " << min_items_per_bin << std::endl;
      total_items += h_histogram[b];
    }
    ASSERT_EQ(params.n_rows, total_items)
      << "Some samples from dataset are either missed of double counted in quantile bins"
      << std::endl;
  }
};

//-------------------------------------------------------------------------------------------------------------------------------------
const std::vector<QuantileTestParameters> inputs = {{1000, 16, 6078587519764079670LLU},
                                                    {1130, 32, 4884670006177930266LLU},
                                                    {1752, 67, 9175325892580481371LLU},
                                                    {2307, 99, 9507819643927052255LLU},
                                                    {5000, 128, 9507819643927052255LLU}};

// float type quantile test
typedef RFQuantileTest<float> RFQuantileTestF;
TEST_P(RFQuantileTestF, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileTestF, ::testing::ValuesIn(inputs));

// double type quantile test
typedef RFQuantileTest<double> RFQuantileTestD;
TEST_P(RFQuantileTestD, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileTestD, ::testing::ValuesIn(inputs));

// float type quantile bins lower bounds test
typedef RFQuantileBinsLowerBoundTest<float> RFQuantileBinsLowerBoundTestF;
TEST_P(RFQuantileBinsLowerBoundTestF, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileBinsLowerBoundTestF, ::testing::ValuesIn(inputs));

// double type quantile bins lower bounds lest
typedef RFQuantileBinsLowerBoundTest<double> RFQuantileBinsLowerBoundTestD;
TEST_P(RFQuantileBinsLowerBoundTestD, test) {}
INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileBinsLowerBoundTestD, ::testing::ValuesIn(inputs));

TEST(RfTest, TextDump)
{
  RF_params rf_params = set_rf_params(2, 2, 1.0, 2, 1, 2, 0.0, true, 1, 1.0, 0, GINI, 1, 128);
  auto forest         = std::make_shared<RandomForestMetaData<float, int>>();

  std::vector<float> X_host      = {1, 2, 3, 6, 7, 8};
  thrust::device_vector<float> X = X_host;
  std::vector<int> y_host        = {0, 0, 1, 1, 1, 0};
  thrust::device_vector<int> y   = y_host;

  raft::handle_t handle(1);
  auto forest_ptr = forest.get();
  fit(handle, forest_ptr, X.data().get(), y.size(), 1, y.data().get(), 2, rf_params);

  std::string expected_start_text = R"(Forest has 1 trees, max_depth 2, and max_leaves 2
Tree #0
 Decision Tree depth --> 1 and n_leaves --> 2
 Tree Fitting - Overall time -->)";

  std::string expected_end_text = R"(└(colid: 0, quesval: 3, best_metric_val: 0.25)
    ├(leaf, prediction: [0.75, 0.25], best_metric_val: 0)
    └(leaf, prediction: [0, 1], best_metric_val: 0))";

  EXPECT_TRUE(get_rf_detailed_text(forest_ptr).find(expected_start_text) != std::string::npos);
  EXPECT_TRUE(get_rf_detailed_text(forest_ptr).find(expected_end_text) != std::string::npos);

  std::string expected_json = R"([
{"nodeid": 0, "split_feature": 0, "split_threshold": 3, "gain": 0.25, "instance_count": 6, "yes": 1, "no": 2, "children": [
  {"nodeid": 1, "leaf_value": [0.75, 0.25], "instance_count": 4},
  {"nodeid": 2, "leaf_value": [0, 1], "instance_count": 2}
]}
])";
  EXPECT_EQ(get_rf_json(forest_ptr), expected_json);
}

}  // end namespace ML
