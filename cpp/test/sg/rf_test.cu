/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <cuda_utils.h>
#include <test_utils.h>
#include "ml_utils.h"
#include "randomforest/randomforest.h"

namespace ML {

using namespace MLCommon;

template<typename T> // template useless for now.
struct RfInputs {
	int n_rows;
	int n_cols;
	int n_trees;
	float max_features;
	float rows_sample;
	int n_inference_rows;
	int max_depth;
	int max_leaves;
	bool bootstrap;
	bool bootstrap_features;
	int n_bins;
	int split_algo;
	int min_rows_per_node;
};

template<typename T>
::std::ostream& operator<<(::std::ostream& os, const RfInputs<T>& dims) {
	return os;
}


template<typename T>
class RfTest: public ::testing::TestWithParam<RfInputs<T> > {
protected:
	void basicTest() {

		params = ::testing::TestWithParam<RfInputs<T>>::GetParam();

		DecisionTree::DecisionTreeParams tree_params(params.max_depth, params.max_leaves, params.max_features, params.n_bins,
							     params.split_algo, params.min_rows_per_node, params.bootstrap_features);
		RF_params rf_params(params.bootstrap, params.bootstrap_features, params.n_trees, params.rows_sample, tree_params);
		//rf_params.print();

		//--------------------------------------------------------
		// Random Forest
		//--------------------------------------------------------

		int data_len = params.n_rows * params.n_cols;
		allocate(data, data_len);
		allocate(labels, params.n_rows);
                hipStream_t stream;
                CUDA_CHECK(hipStreamCreate(&stream) );

		// Populate data (assume Col major)
		std::vector<T> data_h = {30.0, 1.0, 2.0, 0.0, 10.0, 20.0, 10.0, 40.0};
		data_h.resize(data_len);
	    updateDevice(data, data_h.data(), data_len, stream);

		// Populate labels
		labels_h = {0, 1, 0, 4};
		labels_h.resize(params.n_rows);
		preprocess_labels(params.n_rows, labels_h, labels_map);
	    updateDevice(labels, labels_h.data(), params.n_rows, stream);

		rf_classifier = new typename rfClassifier<T>::rfClassifier(rf_params);

		cumlHandle handle;
                handle.setStream(stream);

		fit(handle, rf_classifier, data, params.n_rows, params.n_cols, labels, labels_map.size());

		CUDA_CHECK(hipStreamSynchronize(stream));
		CUDA_CHECK(hipStreamDestroy(stream));

		// Inference data: same as train, but row major
		int inference_data_len = params.n_inference_rows * params.n_cols;
		inference_data_h = {30.0, 10.0, 1.0, 20.0, 2.0, 10.0, 0.0, 40.0};
		inference_data_h.resize(inference_data_len);

		
		// Predict and compare against known labels
		predicted_labels.resize(params.n_inference_rows);
		RF_metrics tmp = cross_validate(handle, rf_classifier, inference_data_h.data(), labels_h.data(),
										params.n_inference_rows, params.n_cols, predicted_labels.data(), false);
		accuracy = tmp.accuracy;
    }

 	void SetUp() override {
		basicTest();
	}

	void TearDown() override {
		accuracy = -1.0f; // reset accuracy
		postprocess_labels(params.n_rows, labels_h, labels_map);
		inference_data_h.clear();
		labels_h.clear();
		labels_map.clear();
		predicted_labels.clear();

		CUDA_CHECK(hipFree(labels));
		CUDA_CHECK(hipFree(data));
		delete rf_classifier;
	}

protected:

	RfInputs<T> params;
	T * data;
    int * labels;
	std::vector<T> inference_data_h;
	std::vector<int> labels_h;
	std::map<int, int> labels_map; //unique map of labels to int vals starting from 0

    rfClassifier<T> * rf_classifier;
	float accuracy = -1.0f; // overriden in each test SetUp and TearDown

	std::vector<int> predicted_labels;
};


const std::vector<RfInputs<float> > inputsf2 = {
	{4, 2, 1, 1.0f, 1.0f, 4, -1, -1, false, false, 4, SPLIT_ALGO::HIST, 2}, // single tree forest, bootstrap false, unlimited depth, 4 bins
	{4, 2, 1, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2},	// single tree forest, bootstrap false, depth of 8, 4 bins
	{4, 2, 10, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2}, //forest with 10 trees, all trees should produce identical predictions (no bootstrapping or column subsampling)
	{4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::HIST, 2}, //forest with 10 trees, with bootstrap and column subsampling enabled, 3 bins
	{4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::GLOBAL_QUANTILE, 2} //forest with 10 trees, with bootstrap and column subsampling enabled, 3 bins, different split algorithm
};

const std::vector<RfInputs<double> > inputsd2 = { // Same as inputsf2
	{4, 2, 1, 1.0f, 1.0f, 4, -1, -1, false, false, 4, SPLIT_ALGO::HIST, 2},
	{4, 2, 1, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2},
	{4, 2, 10, 1.0f, 1.0f, 4, 8, -1, false, false, 4, SPLIT_ALGO::HIST, 2},
	{4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::HIST, 2},
	{4, 2, 10, 0.8f, 0.8f, 4, 8, -1, true, false, 3, SPLIT_ALGO::GLOBAL_QUANTILE, 2}
};


typedef RfTest<float> RfTestF;
TEST_P(RfTestF, Fit) {
	//rf_classifier->print_rf_detailed(); // Prints all trees in the forest. Leaf nodes use the remapped values from labels_map.
	if (!params.bootstrap && (params.max_features == 1.0f)) {
		ASSERT_TRUE(accuracy == 1.0f);
	} else  {
		ASSERT_TRUE(accuracy >= 0.75f); // Empirically derived accuracy range
	}
}

typedef RfTest<double> RfTestD;
TEST_P(RfTestD, Fit) {
	if (!params.bootstrap && (params.max_features == 1.0f)) {
		ASSERT_TRUE(accuracy == 1.0f);
	} else  {
		ASSERT_TRUE(accuracy >= 0.75f);
	}
}

INSTANTIATE_TEST_CASE_P(RfTests, RfTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(RfTests, RfTestD, ::testing::ValuesIn(inputsd2));

} // end namespace ML
