#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/tree/algo_helper.h>
#include <decisiontree/quantile/quantile.h>
#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/transpose.h>
#include <test_utils.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cuml/datasets/make_blobs.hpp>
#include <cuml/ensemble/randomforest.hpp>
#include <memory>
#include <raft/cuda_utils.cuh>
#include <raft/handle.hpp>
#include <random/make_blobs.cuh>
#include <random>
#include <tuple>

namespace ML {

// Utils for changing tuple into struct
namespace detail {
template <typename result_type, typename... types, std::size_t... indices>
result_type make_struct(std::tuple<types...> t,
                        std::index_sequence<indices...>)  // &, &&, const && etc.
{
  return {std::get<indices>(t)...};
}

}  // namespace detail

template <typename result_type, typename... types>
result_type make_struct(std::tuple<types...> t)  // &, &&, const && etc.
{
  return detail::make_struct<result_type, types...>(
    t, std::index_sequence_for<types...>{});  // if there is repeated types, then the change for
                                              // using std::index_sequence_for is trivial
}

template <int I, typename RandomGenT, typename ParamT, typename T>
void SampleWithoutReplacemment(RandomGenT& gen, std::vector<ParamT>& sample, std::vector<T> x)
{
  std::vector<T> parameter_sample(sample.size());
  std::shuffle(x.begin(), x.end(), gen);
  for (size_t i = 0; i < sample.size(); i++) {
    parameter_sample[i] = x[i % x.size()];
  }
  std::shuffle(parameter_sample.begin(), parameter_sample.end(), gen);
  for (size_t i = 0; i < sample.size(); i++) {
    std::get<I>(sample[i]) = parameter_sample[i];
  }
}

template <int I, typename RandomGenT, typename ParamT, typename T, typename... Args>
void AddParameters(RandomGenT& gen, std::vector<ParamT>& sample, std::vector<T> x, Args... args)
{
  SampleWithoutReplacemment<I>(gen, sample, x);
  if constexpr (sizeof...(args) > 0) { AddParameters<I + 1>(gen, sample, args...); }
}

template <typename ParamT, typename... Args>
std::vector<ParamT> SampleParameters(int num_samples, size_t seed, Args... args)
{
  std::vector<typename ParamT::types> tuple_sample(num_samples);
  std::default_random_engine gen(seed);
  AddParameters<0>(gen, tuple_sample, args...);
  std::vector<ParamT> sample(num_samples);
  for (size_t i = 0; i < num_samples; i++) {
    sample[i] = make_struct<ParamT>(tuple_sample[i]);
  }
  return sample;
}

struct RfTestParams {
  int n_rows;
  int n_cols;
  int n_trees;
  float max_features;
  float max_samples;
  int max_depth;
  int max_leaves;
  bool bootstrap;
  int n_bins;
  int min_samples_leaf;
  int min_samples_split;
  float min_impurity_decrease;
  int n_streams;
  CRITERION split_criterion;
  int seed;
  int n_labels;
  bool double_precision;
  // c++ has no reflection, so we enumerate the types here
  // This must be updated if new fields are added
  using types = std::tuple<int,
                           int,
                           int,
                           float,
                           float,
                           int,
                           int,
                           bool,
                           int,
                           int,
                           int,
                           float,
                           int,
                           CRITERION,
                           int,
                           int,
                           bool>;
};

std::ostream& operator<<(std::ostream& os, const RfTestParams& ps)
{
  os << "n_rows = " << ps.n_rows << ", n_cols = " << ps.n_cols;
  os << ", n_trees = " << ps.n_trees << ", max_features = " << ps.max_features;
  os << ", max_samples = " << ps.max_samples << ", max_depth = " << ps.max_depth;
  os << ", max_leaves = " << ps.max_leaves << ", bootstrap = " << ps.bootstrap;
  os << ", n_bins = " << ps.n_bins << ", min_samples_leaf = " << ps.min_samples_leaf;
  os << ", min_samples_split = " << ps.min_samples_split;
  os << ", min_impurity_decrease = " << ps.min_impurity_decrease
     << ", n_streams = " << ps.n_streams;
  os << ", split_criterion = " << ps.split_criterion << ", seed = " << ps.seed;
  os << ", n_labels = " << ps.n_labels << ", double_precision = " << ps.double_precision;
  return os;
}

// Classification
template <typename DataT, typename LabelT, typename RfT>
void TrainScore(const raft::handle_t& handle,
                RfTestParams params,
                DataT* X,
                DataT* X_transpose,
                LabelT* y,
                RfT* forest,
                RF_metrics& metrics)
{
  RF_params rf_params = set_rf_params(params.max_depth,
                                      params.max_leaves,
                                      params.max_features,
                                      params.n_bins,
                                      params.min_samples_leaf,
                                      params.min_samples_split,
                                      params.min_impurity_decrease,
                                      params.bootstrap,
                                      params.n_trees,
                                      params.max_samples,
                                      0,
                                      params.split_criterion,
                                      params.n_streams,
                                      128);
  if constexpr (std::is_integral_v<LabelT>) {
    fit(handle, forest, X, params.n_rows, params.n_cols, y, params.n_labels, rf_params);
  } else {
    fit(handle, forest, X, params.n_rows, params.n_cols, y, rf_params);
  }

  thrust::device_vector<LabelT> pred(params.n_rows);
  predict(handle, forest, X_transpose, params.n_rows, params.n_cols, pred.data().get());
  // Predict and compare against known labels
  metrics = score(handle, forest, y, params.n_rows, pred.data().get());
}

template <typename DataT, typename LabelT>
class RfSpecialisedTest {
 public:
  RfSpecialisedTest(RfTestParams params) : params(params)
  {
    raft::handle_t handle(params.n_streams);
    X.resize(params.n_rows * params.n_cols);
    X_transpose.resize(params.n_rows * params.n_cols);
    y.resize(params.n_rows);
    // Make data
    if constexpr (std::is_integral<LabelT>::value) {
      Datasets::make_blobs(handle,
                           X.data().get(),
                           y.data().get(),
                           params.n_rows,
                           params.n_cols,
                           params.n_labels,
                           false,
                           nullptr,
                           nullptr,
                           5.0,
                           false,
                           -10.0f,
                           10.0f,
                           params.seed);
    } else {
      thrust::device_vector<int> y_temp(params.n_rows);
      Datasets::make_blobs(handle,
                           X.data().get(),
                           y_temp.data().get(),
                           params.n_rows,
                           params.n_cols,
                           params.n_labels,
                           false,
                           nullptr,
                           nullptr,
                           5.0,
                           false,
                           -10.0f,
                           10.0f,
                           params.seed);
      // if regression, make the labels normally distributed
      raft::random::Rng r(4);
      thrust::device_vector<double> normal(params.n_rows);
      r.normal(normal.data().get(), normal.size(), 0.0, 2.0, nullptr);
      thrust::transform(
        normal.begin(), normal.end(), y_temp.begin(), y.begin(), thrust::plus<LabelT>());
    }
    raft::linalg::transpose(
      handle, X.data().get(), X_transpose.data().get(), params.n_rows, params.n_cols, nullptr);
    forest.reset(new typename ML::RandomForestMetaData<DataT, LabelT>);
    TrainScore(handle,
               params,
               X.data().get(),
               X_transpose.data().get(),
               y.data().get(),
               forest.get(),
               training_metrics);

    Test();
  }
  // Current model should be at least as accurate as a model with depth - 1
  void TestAccuracyImprovement()
  {
    if (params.max_depth <= 1) { return; }
    // accuracy is not guaranteed to improve with bootstrapping
    if (params.bootstrap) { return; }
    raft::handle_t handle(params.n_streams);
    RfTestParams alt_params = params;
    alt_params.max_depth--;
    RF_metrics alt_metrics;
    ML::RandomForestMetaData<DataT, LabelT> alt_forest;
    TrainScore(handle,
               alt_params,
               X.data().get(),
               X_transpose.data().get(),
               y.data().get(),
               &alt_forest,
               alt_metrics);
    double eps = 1e-8;
    if (params.split_criterion == MSE) {
      EXPECT_LE(training_metrics.mean_squared_error, alt_metrics.mean_squared_error + eps);
    } else if (params.split_criterion == MAE) {
      EXPECT_LE(training_metrics.mean_abs_error, alt_metrics.mean_abs_error + eps);
    } else {
      EXPECT_GE(training_metrics.accuracy, alt_metrics.accuracy);
    }
  }
  // Regularisation parameters are working correctly
  void TestTreeSize()
  {
    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      EXPECT_LE(forest->trees[i].depth_counter, params.max_depth);
      if (params.max_leaves > 0) { EXPECT_LE(forest->trees[i].leaf_counter, params.max_leaves); }
      EXPECT_LE(forest->trees[i].leaf_counter,
                raft::ceildiv(params.n_rows, params.min_samples_leaf));
    }
  }
  void TestDeterminism()
  {
    // Regression models use floating point atomics, so are not bitwise reproducible
    bool is_regression = params.split_criterion == MSE || params.split_criterion == MAE;
    if (is_regression) return;

    // Repeat training
    RF_metrics metrics;
    raft::handle_t handle(params.n_streams);
    ML::RandomForestMetaData<DataT, LabelT> alt_forest;
    TrainScore(handle,
               params,
               X.data().get(),
               X_transpose.data().get(),
               y.data().get(),
               &alt_forest,
               metrics);

    for (int i = 0u; i < forest->rf_params.n_trees; i++) {
      EXPECT_EQ(forest->trees[i].sparsetree, alt_forest.trees[i].sparsetree);
    }
  }
  void Test()
  {
    TestAccuracyImprovement();
    // Bugs
    // TestDeterminism();
    // TestTreeSize();
  }

  RF_metrics training_metrics;
  thrust::device_vector<DataT> X;
  thrust::device_vector<DataT> X_transpose;
  thrust::device_vector<LabelT> y;
  RfTestParams params;
  std::shared_ptr<RandomForestMetaData<DataT, LabelT>> forest;
};

// Dispatch tests based on any template parameters
class RfTest : public ::testing::TestWithParam<RfTestParams> {
 public:
  void SetUp() override
  {
    RfTestParams params = ::testing::TestWithParam<RfTestParams>::GetParam();
    bool is_regression  = params.split_criterion == MSE || params.split_criterion == MAE;
    if (params.double_precision) {
      if (is_regression) {
        RfSpecialisedTest<double, double> test(params);
      } else {
        RfSpecialisedTest<double, int> test(params);
      }
    } else {
      if (is_regression) {
        RfSpecialisedTest<float, float> test(params);
      } else {
        RfSpecialisedTest<float, int> test(params);
      }
    }
  }
};

TEST_P(RfTest, PropertyBasedTest) {}

// Parameter ranges to test
std::vector<int> n_rows         = {10, 100, 1452};
std::vector<int> n_cols         = {1, 5, 152, 1014};
std::vector<int> n_trees        = {1, 5, 17};
std::vector<float> max_features = {0.1f, 0.5f, 1.0f};
std::vector<float> max_samples  = {0.1f, 0.5f, 1.0f};
std::vector<int> max_depth      = {1, 10, 30};
std::vector<int> max_leaves = {-1};  // Bug for max_leaves, non-determinism as threads compete to
                                     // place their nodes inside this limit
std::vector<bool> bootstrap = {false, true};
std::vector<int> n_bins     = {2, 57, 128};  // Bug for n_bins > 128. Uses too much shared memory.
std::vector<int> min_samples_leaf        = {1, 10, 30};
std::vector<int> min_samples_split       = {2, 10};
std::vector<float> min_impurity_decrease = {0.0, 1.0f, 10.0f};
std::vector<int> n_streams               = {1, 2, 10};
std::vector<CRITERION> split_criterion   = {CRITERION::MSE, CRITERION::GINI, CRITERION::ENTROPY};
std::vector<int> seed                    = {0, 17};
std::vector<int> n_labels                = {2, 10, 30};
std::vector<bool> double_precision       = {false, true};

int n_tests = 100;

INSTANTIATE_TEST_CASE_P(RfTests,
                        RfTest,
                        ::testing::ValuesIn(SampleParameters<RfTestParams>(n_tests,
                                                                           0,
                                                                           n_rows,
                                                                           n_cols,
                                                                           n_trees,
                                                                           max_features,
                                                                           max_samples,
                                                                           max_depth,
                                                                           max_leaves,
                                                                           bootstrap,
                                                                           n_bins,
                                                                           min_samples_leaf,
                                                                           min_samples_split,
                                                                           min_impurity_decrease,
                                                                           n_streams,
                                                                           split_criterion,
                                                                           seed,
                                                                           n_labels,
                                                                           double_precision)));

struct QuantileTestParameters {
  int n_rows;
  int n_bins;
  uint64_t seed;
};

template <typename T>
class RFQuantileTest : public ::testing::TestWithParam<QuantileTestParameters> {
 public:
  void SetUp() override
  {
    auto params = ::testing::TestWithParam<QuantileTestParameters>::GetParam();

    thrust::device_vector<T> data(params.n_rows);
    thrust::device_vector<T> quantiles(params.n_bins);
    thrust::device_vector<int> histogram(params.n_bins);
    thrust::host_vector<int> h_histogram(params.n_bins);

    raft::random::Rng r(8);
    r.normal(data.data().get(), data.size(), T(0.0), T(2.0), nullptr);
    raft::handle_t handle;
    DT::computeQuantiles(quantiles.data().get(),
                         params.n_bins,
                         data.data().get(),
                         params.n_rows,
                         1,
                         handle.get_device_allocator(),
                         nullptr);

    auto d_quantiles = quantiles.data();
    auto d_histogram = histogram.data().get();
    thrust::for_each(data.begin(), data.end(), [=] __device__(T x) {
      for (int j = 0; j < params.n_bins; j++) {
        if (x <= d_quantiles[j]) {
          atomicAdd(&d_histogram[j], 1);
          break;
        }
      }
    });

    h_histogram           = histogram;
    int max_items_per_bin = raft::ceildiv(params.n_rows, params.n_bins);
    int min_items_per_bin = max_items_per_bin - 1;
    int total_items       = 0;
    for (int b = 0; b < params.n_bins; b++) {
      ASSERT_TRUE(h_histogram[b] == max_items_per_bin || h_histogram[b] == min_items_per_bin)
        << "No. samples in bin[" << b << "] = " << h_histogram[b] << " Expected "
        << max_items_per_bin << " or " << min_items_per_bin << std::endl;
      total_items += h_histogram[b];
    }
    ASSERT_EQ(params.n_rows, total_items)
      << "Some samples from dataset are either missed of double counted in quantile bins"
      << std::endl;
  }
};

//-------------------------------------------------------------------------------------------------------------------------------------
const std::vector<QuantileTestParameters> inputs = {{1000, 16, 6078587519764079670LLU},
                                                    {1130, 32, 4884670006177930266LLU},
                                                    {1752, 67, 9175325892580481371LLU},
                                                    {2307, 99, 9507819643927052255LLU},
                                                    {5000, 128, 9507819643927052255LLU}};

typedef RFQuantileTest<float> RFQuantileTestF;
TEST_P(RFQuantileTestF, test) {}

INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileTestF, ::testing::ValuesIn(inputs));

typedef RFQuantileTest<double> RFQuantileTestD;
TEST_P(RFQuantileTestD, test) {}

INSTANTIATE_TEST_CASE_P(RfTests, RFQuantileTestD, ::testing::ValuesIn(inputs));
}  // end namespace ML
