/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "knn_test_helper.cuh"

namespace ML {
namespace KNN {
namespace opg {

template <>
void generate_partitions(float *data, float *outputs, size_t n_rows, int n_cols,
                         int n_clusters, int my_rank,
                         std::shared_ptr<deviceAllocator> allocator,
                         hipStream_t stream) {
  Random::make_blobs<float, int>(data, (int *)outputs, (int)n_rows, (int)n_cols,
                                 n_clusters, allocator, stream, true, nullptr,
                                 nullptr, 1.0, -10.0, 10.0, my_rank);
  MLCommon::LinAlg::convert_array(outputs, (int *)outputs, n_rows, stream);
}

class KNNRegressTest : public ::testing::TestWithParam<KNNParams> {
 public:
  bool runTest(const KNNParams &params) {
    KNNTestHelper<float> knn_th;
    knn_th.generate_data(params);

    /**
     * Execute knn_regress()
     */
    knn_regress(knn_th.handle, &(knn_th.out_parts), &(knn_th.out_i_parts),
                &(knn_th.out_d_parts), knn_th.index_parts, *(knn_th.idx_desc),
                knn_th.query_parts, *(knn_th.query_desc), knn_th.y, false,
                false, params.k, params.n_outputs, params.batch_size, true);

    knn_th.display_results();
    knn_th.release_ressources(params);

    int actual = 1;
    int expected = 1;
    return CompareApprox<int>(1)(actual, expected);
  }
};

const std::vector<KNNParams> inputs = {{5, 1, 8, 50, 3, 2, 2, 12}};

typedef KNNRegressTest KNNReTest;

TEST_P(KNNReTest, Result) { ASSERT_TRUE(runTest(GetParam())); }

INSTANTIATE_TEST_CASE_P(KNNRegressTest, KNNReTest, ::testing::ValuesIn(inputs));

}  // namespace opg
}  // namespace KNN
}  // namespace ML
