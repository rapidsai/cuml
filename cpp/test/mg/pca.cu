/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <raft/linalg/cublas_wrappers.h>
#include <test_utils.h>
#include <cuda_utils.cuh>
#include <cuml/common/logger.hpp>
#include <cuml/decomposition/pca_mg.hpp>
#include <matrix/matrix.cuh>
#include <opg/linalg/gemm.hpp>
#include <opg/matrix/matrix_utils.hpp>
#include "test_opg_utils.h"

namespace MLCommon {
namespace Test {
namespace opg {

struct PCAOpgParams {
  int M;
  int N;
  int N_components;
  ML::mg_solver algorithm;
  std::vector<int> partSizes;
  std::vector<int> ranksOwners;
  Matrix::Layout layout;
  unsigned long long int seed;
};

template <typename T>
class PCAOpgTest : public testing::TestWithParam<PCAOpgParams> {
 public:
  void SetUp() {
    params = GetParam();
    handle = new raft::handle_t();
    ML::initialize_mpi_comms(*handle, MPI_COMM_WORLD);

    // Prepare resource
    const raft::handle_t& h = handle;
    const cumlCommunicator& comm = h.getCommunicator();
    stream = h.get_stream();
    const auto allocator = h.get_device_allocator();
    hipblasHandle_t cublasHandle = h.get_cublas_handle();

    myRank = comm.getRank();
    totalRanks = comm.getSize();
    Random::Rng r(params.seed + myRank);

    CUBLAS_CHECK(hipblasSetStream(cublasHandle, stream));

    if (myRank == 0) {
      std::cout << "Testing PCA of " << params.M << " x " << params.N
                << " matrix" << std::endl;
    }

    // Prepare X matrix
    std::vector<Matrix::RankSizePair*> totalPartsToRanks;
    for (int i = 0; i < params.partSizes.size(); i++) {
      Matrix::RankSizePair* rspt = new Matrix::RankSizePair(
        params.ranksOwners[i] % totalRanks, params.partSizes[i]);
      totalPartsToRanks.push_back(rspt);
    }
    Matrix::PartDescriptor desc(params.M, params.N, totalPartsToRanks,
                                comm.getRank(), params.layout);
    std::vector<Matrix::Data<T>*> inParts;
    Matrix::opg::allocate(h, inParts, desc, myRank, stream);
    Matrix::opg::randomize(h, r, inParts, desc, myRank, stream, T(10.0),
                           T(20.0));
    h.waitOnUserStream();

    prmsPCA.n_rows = params.M;
    prmsPCA.n_cols = params.N;
    prmsPCA.n_components = params.N_components;
    prmsPCA.whiten = false;
    prmsPCA.n_iterations = 100;
    prmsPCA.tol = 0.01;
    prmsPCA.algorithm = params.algorithm;

    device_buffer<T> components(allocator, stream,
                                prmsPCA.n_components * prmsPCA.n_cols);

    device_buffer<T> explained_var(allocator, stream, prmsPCA.n_components);

    device_buffer<T> explained_var_ratio(allocator, stream,
                                         prmsPCA.n_components);

    device_buffer<T> singular_vals(allocator, stream, prmsPCA.n_components);

    device_buffer<T> mu(allocator, stream, prmsPCA.n_cols);

    device_buffer<T> noise_vars(allocator, stream, prmsPCA.n_components);

    ML::PCA::opg::fit(*handle, inParts, desc, components.data(),
                      explained_var.data(), explained_var_ratio.data(),
                      singular_vals.data(), mu.data(), noise_vars.data(),
                      prmsPCA, false);

    CUML_LOG_DEBUG(MLCommon::arr2Str(singular_vals.data(), params.N_components,
                                     "Singular Vals", stream)
                     .c_str());

    CUML_LOG_DEBUG(MLCommon::arr2Str(explained_var.data(), params.N_components,
                                     "Explained Variance", stream)
                     .c_str());

    CUML_LOG_DEBUG(MLCommon::arr2Str(explained_var_ratio.data(),
                                     params.N_components,
                                     "Explained Variance Ratio", stream)
                     .c_str());

    CUML_LOG_DEBUG(MLCommon::arr2Str(components.data(),
                                     params.N_components * params.N,
                                     "Components", stream)
                     .c_str());

    Matrix::opg::deallocate(h, inParts, desc, myRank, stream);
  }

  void TearDown() { delete handle; }

 protected:
  PCAOpgParams params;
  raft::handle_t* handle;
  hipStream_t stream;
  int myRank;
  int totalRanks;
  ML::paramsPCAMG prmsPCA;
};

const std::vector<PCAOpgParams> inputs = {{20,
                                           4,
                                           2,
                                           ML::mg_solver::COV_EIG_JACOBI,
                                           {11, 9},
                                           {1, 0},
                                           Matrix::LayoutColMajor,
                                           223548ULL},
                                          {20,
                                           4,
                                           2,
                                           ML::mg_solver::COV_EIG_DQ,
                                           {11, 9},
                                           {1, 0},
                                           Matrix::LayoutColMajor,
                                           223548ULL},
                                          {20,
                                           4,
                                           2,
                                           ML::mg_solver::QR,
                                           {11, 9},
                                           {1, 0},
                                           Matrix::LayoutColMajor,
                                           223548ULL}};

typedef PCAOpgTest<float> PCAOpgTestF;

TEST_P(PCAOpgTestF, Result) {
  if (myRank == 0) {
    // We should be inverse transforming and checking against the original
    // data here. Github reference: https://github.com/rapidsai/cuml/issues/2474

    ASSERT_TRUE(true);
  }
}

INSTANTIATE_TEST_CASE_P(PCAOpgTest, PCAOpgTestF, ::testing::ValuesIn(inputs));

typedef PCAOpgTest<double> PCAOpgTestD;

TEST_P(PCAOpgTestD, Result) {
  if (myRank == 0) {
    // We should be inverse transforming and checking against the original
    // data here. Github reference: https://github.com/rapidsai/cuml/issues/2474

    ASSERT_TRUE(true);
  }
}

INSTANTIATE_TEST_CASE_P(PCAOpgTest, PCAOpgTestD, ::testing::ValuesIn(inputs));

}  // end namespace opg
}  // end namespace Test
}  // end namespace MLCommon