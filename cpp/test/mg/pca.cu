/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/cublas_wrappers.h>
#include <test_utils.h>
#include <cuml/common/device_buffer.hpp>
#include <cuml/common/logger.hpp>
#include <cuml/decomposition/pca_mg.hpp>
#include <opg/linalg/gemm.hpp>
#include <opg/matrix/matrix_utils.hpp>
#include <raft/cuda_utils.cuh>
#include <raft/matrix/matrix.cuh>
#include "test_opg_utils.h"

#include <raft/comms/mpi_comms.hpp>

namespace MLCommon {
namespace Test {
namespace opg {

struct PCAOpgParams {
  int M;
  int N;
  int N_components;
  ML::mg_solver algorithm;
  std::vector<int> partSizes;
  std::vector<int> ranksOwners;
  Matrix::Layout layout;
  unsigned long long int seed;
};

template <typename T>
class PCAOpgTest : public testing::TestWithParam<PCAOpgParams> {
 public:
  void SetUp() {
    params = GetParam();
    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);

    // Prepare resource

    const raft::comms::comms_t& comm = handle.get_comms();
    stream = handle.get_stream();
    const auto allocator = handle.get_device_allocator();
    hipblasHandle_t cublasHandle = handle.get_cublas_handle();

    myRank = comm.get_rank();
    totalRanks = comm.get_size();
    raft::random::Rng r(params.seed + myRank);

    CUBLAS_CHECK(hipblasSetStream(cublasHandle, stream));

    if (myRank == 0) {
      std::cout << "Testing PCA of " << params.M << " x " << params.N
                << " matrix" << std::endl;
    }

    // Prepare X matrix
    std::vector<Matrix::RankSizePair*> totalPartsToRanks;
    for (int i = 0; i < params.partSizes.size(); i++) {
      Matrix::RankSizePair* rspt = new Matrix::RankSizePair(
        params.ranksOwners[i] % totalRanks, params.partSizes[i]);
      totalPartsToRanks.push_back(rspt);
    }
    Matrix::PartDescriptor desc(params.M, params.N, totalPartsToRanks,
                                comm.get_rank(), params.layout);
    std::vector<Matrix::Data<T>*> inParts;
    Matrix::opg::allocate(handle, inParts, desc, myRank, stream);
    Matrix::opg::randomize(handle, r, inParts, desc, myRank, stream, T(10.0),
                           T(20.0));
    handle.wait_on_user_stream();

    prmsPCA.n_rows = params.M;
    prmsPCA.n_cols = params.N;
    prmsPCA.n_components = params.N_components;
    prmsPCA.whiten = false;
    prmsPCA.n_iterations = 100;
    prmsPCA.tol = 0.01;
    prmsPCA.algorithm = params.algorithm;

    device_buffer<T> components(allocator, stream,
                                prmsPCA.n_components * prmsPCA.n_cols);

    device_buffer<T> explained_var(allocator, stream, prmsPCA.n_components);

    device_buffer<T> explained_var_ratio(allocator, stream,
                                         prmsPCA.n_components);

    device_buffer<T> singular_vals(allocator, stream, prmsPCA.n_components);

    device_buffer<T> mu(allocator, stream, prmsPCA.n_cols);

    device_buffer<T> noise_vars(allocator, stream, prmsPCA.n_components);

    ML::PCA::opg::fit(handle, inParts, desc, components.data(),
                      explained_var.data(), explained_var_ratio.data(),
                      singular_vals.data(), mu.data(), noise_vars.data(),
                      prmsPCA, false);

    CUML_LOG_DEBUG(raft::arr2Str(singular_vals.data(), params.N_components,
                                 "Singular Vals", stream)
                     .c_str());

    CUML_LOG_DEBUG(raft::arr2Str(explained_var.data(), params.N_components,
                                 "Explained Variance", stream)
                     .c_str());

    CUML_LOG_DEBUG(raft::arr2Str(explained_var_ratio.data(),
                                 params.N_components,
                                 "Explained Variance Ratio", stream)
                     .c_str());

    CUML_LOG_DEBUG(raft::arr2Str(components.data(),
                                 params.N_components * params.N, "Components",
                                 stream)
                     .c_str());

    Matrix::opg::deallocate(handle, inParts, desc, myRank, stream);
  }

 protected:
  PCAOpgParams params;
  raft::handle_t handle;
  hipStream_t stream;
  int myRank;
  int totalRanks;
  ML::paramsPCAMG prmsPCA;
};

const std::vector<PCAOpgParams> inputs = {{20,
                                           4,
                                           2,
                                           ML::mg_solver::COV_EIG_JACOBI,
                                           {11, 9},
                                           {1, 0},
                                           Matrix::LayoutColMajor,
                                           223548ULL},
                                          {20,
                                           4,
                                           2,
                                           ML::mg_solver::COV_EIG_DQ,
                                           {11, 9},
                                           {1, 0},
                                           Matrix::LayoutColMajor,
                                           223548ULL},
                                          {20,
                                           4,
                                           2,
                                           ML::mg_solver::QR,
                                           {11, 9},
                                           {1, 0},
                                           Matrix::LayoutColMajor,
                                           223548ULL}};

typedef PCAOpgTest<float> PCAOpgTestF;

TEST_P(PCAOpgTestF, Result) {
  if (myRank == 0) {
    // We should be inverse transforming and checking against the original
    // data here. Github reference: https://github.com/rapidsai/cuml/issues/2474

    ASSERT_TRUE(true);
  }
}

INSTANTIATE_TEST_CASE_P(PCAOpgTest, PCAOpgTestF, ::testing::ValuesIn(inputs));

typedef PCAOpgTest<double> PCAOpgTestD;

TEST_P(PCAOpgTestD, Result) {
  if (myRank == 0) {
    // We should be inverse transforming and checking against the original
    // data here. Github reference: https://github.com/rapidsai/cuml/issues/2474

    ASSERT_TRUE(true);
  }
}

INSTANTIATE_TEST_CASE_P(PCAOpgTest, PCAOpgTestD, ::testing::ValuesIn(inputs));

}  // end namespace opg
}  // end namespace Test
}  // end namespace MLCommon
