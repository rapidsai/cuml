/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <nccl.h>
#include <raft/comms/std_comms.hpp>
#include <raft/core/handle.hpp>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>
#include <rmm/device_uvector.hpp>
#include <stdio.h>
#include <test_utils.h>
#include <vector>

#include <cuml/cluster/kmeans.hpp>
#include <cuml/cluster/kmeans_mg.hpp>
#include <cuml/common/logger.hpp>
#include <cuml/datasets/make_blobs.hpp>
#include <cuml/metrics/metrics.hpp>
#include <thrust/fill.h>

#define NCCLCHECK(cmd)                                                                        \
  do {                                                                                        \
    ncclResult_t res = cmd;                                                                   \
    if (res != ncclSuccess) {                                                                 \
      printf("Failed, NCCL error %s:%d '%s'\n", __FILE__, __LINE__, ncclGetErrorString(res)); \
      exit(EXIT_FAILURE);                                                                     \
    }                                                                                         \
  } while (0)

namespace ML {

using namespace Datasets;
using namespace Metrics;

template <typename T>
struct KmeansInputs {
  int n_row;
  int n_col;
  int n_clusters;
  T tol;
  bool weighted;
};

template <typename T>
class KmeansTest : public ::testing::TestWithParam<KmeansInputs<T>> {
 protected:
  KmeansTest()
    : stream(handle.get_stream()),
      d_labels(0, stream),
      d_labels_ref(0, stream),
      d_centroids(0, stream),
      d_sample_weight(0, stream)
  {
  }

  void basicTest()
  {
    testparams = ::testing::TestWithParam<KmeansInputs<T>>::GetParam();
    ncclComm_t nccl_comm;
    NCCLCHECK(ncclCommInitAll(&nccl_comm, 1, {0}));
    raft::comms::build_comms_nccl_only(&handle, nccl_comm, 1, 0);

    int n_samples              = testparams.n_row;
    int n_features             = testparams.n_col;
    params.n_clusters          = testparams.n_clusters;
    params.tol                 = testparams.tol;
    params.n_init              = 5;
    params.rng_state.seed      = 1;
    params.oversampling_factor = 1;

    auto stream = handle.get_stream();
    rmm::device_uvector<T> X(n_samples * n_features, stream);
    rmm::device_uvector<int> labels(n_samples, stream);

    make_blobs(handle,
               X.data(),
               labels.data(),
               n_samples,
               n_features,
               params.n_clusters,
               true,
               nullptr,
               nullptr,
               1.0,
               false,
               -10.0f,
               10.0f,
               1234ULL);

    d_labels.resize(n_samples, stream);
    d_labels_ref.resize(n_samples, stream);
    d_centroids.resize(params.n_clusters * n_features, stream);

    T* d_sample_weight_ptr = nullptr;
    if (testparams.weighted) {
      d_sample_weight.resize(n_samples, stream);
      d_sample_weight_ptr = d_sample_weight.data();
      thrust::fill(
        thrust::cuda::par.on(stream), d_sample_weight_ptr, d_sample_weight_ptr + n_samples, 1);
    }

    raft::copy(d_labels_ref.data(), labels.data(), n_samples, stream);

    handle.sync_stream(stream);

    T inertia  = 0;
    int n_iter = 0;

    ML::kmeans::opg::fit(handle,
                         params,
                         X.data(),
                         n_samples,
                         n_features,
                         d_sample_weight_ptr,
                         d_centroids.data(),
                         inertia,
                         n_iter);

    kmeans::predict(handle,
                    params,
                    d_centroids.data(),
                    X.data(),
                    n_samples,
                    n_features,
                    d_sample_weight_ptr,
                    true,
                    d_labels.data(),
                    inertia);

    score = adjusted_rand_index(handle, d_labels_ref.data(), d_labels.data(), n_samples);
    handle.sync_stream(stream);

    if (score < 0.99) {
      std::stringstream ss;
      ss << "Expected: " << raft::arr2Str(d_labels_ref.data(), 25, "d_labels_ref", stream);
      CUML_LOG_WARN(ss.str().c_str());
      ss.str(std::string());
      ss << "Actual: " << raft::arr2Str(d_labels.data(), 25, "d_labels", stream);
      CUML_LOG_WARN(ss.str().c_str());
      CUML_LOG_WARN("Score = %lf", score);
    }

    ncclCommDestroy(nccl_comm);
  }

  void SetUp() override { basicTest(); }

 protected:
  raft::handle_t handle;
  hipStream_t stream;
  KmeansInputs<T> testparams;
  rmm::device_uvector<int> d_labels;
  rmm::device_uvector<int> d_labels_ref;
  rmm::device_uvector<T> d_centroids;
  rmm::device_uvector<T> d_sample_weight;
  double score;
  ML::kmeans::KMeansParams params;
};

const std::vector<KmeansInputs<float>> inputsf2 = {{1000, 32, 5, 0.0001, true},
                                                   {1000, 32, 5, 0.0001, false},
                                                   {1000, 100, 20, 0.0001, true},
                                                   {1000, 100, 20, 0.0001, false},
                                                   {10000, 32, 10, 0.0001, true},
                                                   {10000, 32, 10, 0.0001, false},
                                                   {10000, 100, 50, 0.0001, true},
                                                   {10000, 100, 50, 0.0001, false}};

const std::vector<KmeansInputs<double>> inputsd2 = {{1000, 32, 5, 0.0001, true},
                                                    {1000, 32, 5, 0.0001, false},
                                                    {1000, 100, 20, 0.0001, true},
                                                    {1000, 100, 20, 0.0001, false},
                                                    {10000, 32, 10, 0.0001, true},
                                                    {10000, 32, 10, 0.0001, false},
                                                    {10000, 100, 50, 0.0001, true},
                                                    {10000, 100, 50, 0.0001, false}};

typedef KmeansTest<float> KmeansTestF;
TEST_P(KmeansTestF, Result) { ASSERT_TRUE(score >= 0.99); }

typedef KmeansTest<double> KmeansTestD;
TEST_P(KmeansTestD, Result) { ASSERT_TRUE(score >= 0.99); }

INSTANTIATE_TEST_CASE_P(KmeansTests, KmeansTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(KmeansTests, KmeansTestD, ::testing::ValuesIn(inputsd2));

}  // end namespace ML