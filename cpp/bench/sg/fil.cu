#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/fil/fil.h>

#include <cuml/tree/algo_helper.h>
#include <treelite/c_api.h>
#include <treelite/tree.h>
#include <cuml/common/logger.hpp>
#include <cuml/cuml.hpp>
#include <cuml/ensemble/randomforest.hpp>
#include <utility>
#include "benchmark.cuh"

namespace ML {
namespace Bench {
namespace fil {

struct Params {
  DatasetParams data;
  RegressionParams blobs;
  ModelHandle model;
  ML::fil::storage_type_t storage;
  ML::fil::algo_t algo;
  RF_params rf;
  int predict_repetitions;
};

class FIL : public RegressionFixture<float> {
  typedef RegressionFixture<float> Base;

 public:
  FIL(const std::string& name, const Params& p)
  /*
        fitting to linear combinations in "y" normally yields trees that check
        values of all significant columns, as well as their linear
        combinations in "X". During inference, the exact threshold
        values do not affect speed. The distribution of column popularity does
        not affect speed barring lots of uninformative columns in succession.
        Hence, this method represents real datasets well enough for both
        classification and regression.
      */
  : RegressionFixture<float>(name, p.data, p.blobs),
    model(p.model),
    p_rest(p) {}

  static void regression_to_classification(float* y, int nrows, int nclasses,
                                           hipStream_t stream) {
    raft::linalg::unaryOp(
      y, y, nrows,
      [=] __device__(float a) {
        return float(lroundf(fabsf(a) * 1000. * nclasses) % nclasses);
      },
      stream);
  }

 protected:
  void runBenchmark(::benchmark::State& state) override {
    if (!params.rowMajor) {
      state.SkipWithError("FIL only supports row-major inputs");
    }
    if (params.nclasses > 1) {
      // convert regression ranges into [0..nclasses-1]
      regression_to_classification(data.y, params.nrows, params.nclasses,
                                   stream);
    }
    // create model
    ML::RandomForestRegressorF rf_model;
    auto* mPtr = &rf_model;
    mPtr->trees = nullptr;
    size_t train_nrows = std::min(params.nrows, 1000);
    fit(*handle, mPtr, data.X, train_nrows, params.ncols, data.y, p_rest.rf);
    CUDA_CHECK(hipStreamSynchronize(stream));

    ML::build_treelite_forest(&model, &rf_model, params.ncols,
                              params.nclasses > 1 ? 2 : 1);
    ML::fil::treelite_params_t tl_params = {
      .algo = p_rest.algo,
      .output_class = params.nclasses > 1,  // cuML RF forest
      .threshold = 1.f / params.nclasses,   //Fixture::DatasetParams
      .storage_type = p_rest.storage};
    ML::fil::from_treelite(*handle, &forest, model, &tl_params);

    // only time prediction
    this->loopOnState(state, [this]() {
      // Dataset<D, L> allocates y assuming one output value per input row,
      // so not supporting predict_proba yet
      for (int i = 0; i < p_rest.predict_repetitions; i++) {
        ML::fil::predict(*this->handle, this->forest, this->data.y,
                         this->data.X, this->params.nrows, false);
      }
    });
  }

  void allocateBuffers(const ::benchmark::State& state) override {
    Base::allocateBuffers(state);
  }

  void deallocateBuffers(const ::benchmark::State& state) override {
    ML::fil::free(*handle, forest);
    Base::deallocateBuffers(state);
  }

 private:
  ML::fil::forest_t forest;
  ModelHandle model;
  Params p_rest;
};

struct FilBenchParams {
  int nrows;
  int ncols;
  int nclasses;
  int max_depth;
  int ntrees;
  ML::fil::storage_type_t storage;
  ML::fil::algo_t algo;
};

std::vector<Params> getInputs() {
  std::vector<Params> out;
  Params p;
  p.data.rowMajor = true;
  p.blobs = {
    .n_informative = -1,   // Just a placeholder value, anyway changed below
    .effective_rank = -1,  // Just a placeholder value, anyway changed below
    .bias = 0.f,
    .tail_strength = 0.1,
    .noise = 0.01,
    .shuffle = false,
    .seed = 12345ULL};

  p.rf = set_rf_params(10,                 /*max_depth */
                       (1 << 20),          /* max_leaves */
                       1.f,                /* max_features */
                       32,                 /* n_bins */
                       1,                  /* split_algo */
                       3,                  /* min_samples_leaf */
                       3,                  /* min_samples_split */
                       0.0f,               /* min_impurity_decrease */
                       true,               /* bootstrap_features */
                       true,               /* bootstrap */
                       1,                  /* n_trees */
                       1.f,                /* max_samples */
                       1234ULL,            /* seed */
                       ML::CRITERION::MSE, /* split_criterion */
                       false,              /* quantile_per_tree */
                       8,                  /* n_streams */
                       false,              /* use_experimental_backend */
                       128                 /* max_batch_size */
  );

  using ML::fil::algo_t;
  using ML::fil::storage_type_t;
  std::vector<FilBenchParams> var_params = {
    {(int)1e6, 20, 1, 5, 1000, storage_type_t::DENSE, algo_t::BATCH_TREE_REORG},
    {(int)1e6, 20, 2, 5, 1000, storage_type_t::DENSE,
     algo_t::BATCH_TREE_REORG}};
  for (auto& i : var_params) {
    p.data.nrows = i.nrows;
    p.data.ncols = i.ncols;
    p.blobs.n_informative = i.ncols / 3;
    p.blobs.effective_rank = i.ncols / 3;
    p.data.nclasses = i.nclasses;
    p.rf.tree_params.max_depth = i.max_depth;
    p.rf.n_trees = i.ntrees;
    p.storage = i.storage;
    p.algo = i.algo;
    p.predict_repetitions = 10;
    out.push_back(p);
  }
  return out;
}

ML_BENCH_REGISTER(Params, FIL, "", getInputs());

}  // end namespace fil
}  // end namespace Bench
}  // end namespace ML
