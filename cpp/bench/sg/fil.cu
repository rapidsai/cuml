#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/fil/fil.h>

#include <cuml/tree/algo_helper.h>
#include <treelite/c_api.h>
#include <treelite/tree.h>
#include <cuml/common/logger.hpp>
#include <cuml/ensemble/randomforest.hpp>
#include <utility>
#include "benchmark.cuh"

namespace ML {
namespace Bench {
namespace fil {

struct Params {
  DatasetParams data;
  RegressionParams blobs;
  ModelHandle model;
  ML::fil::storage_type_t storage;
  ML::fil::algo_t algo;
  RF_params rf;
  int predict_repetitions;
};

class FIL : public RegressionFixture<float> {
  typedef RegressionFixture<float> Base;

 public:
  FIL(const std::string& name, const Params& p)
  /*
        fitting to linear combinations in "y" normally yields trees that check
        values of all significant columns, as well as their linear
        combinations in "X". During inference, the exact threshold
        values do not affect speed. The distribution of column popularity does
        not affect speed barring lots of uninformative columns in succession.
        Hence, this method represents real datasets well enough for both
        classification and regression.
      */
  : RegressionFixture<float>(name, p.data, p.blobs), model(p.model), p_rest(p)
  {
  }

  static void regression_to_classification(float* y, int nrows, int nclasses, hipStream_t stream)
  {
    raft::linalg::unaryOp(
      y,
      y,
      nrows,
      [=] __device__(float a) { return float(lroundf(fabsf(a) * 1000. * nclasses) % nclasses); },
      stream);
  }

 protected:
  void runBenchmark(::benchmark::State& state) override
  {
    if (!params.rowMajor) { state.SkipWithError("FIL only supports row-major inputs"); }
    if (params.nclasses > 1) {
      // convert regression ranges into [0..nclasses-1]
      regression_to_classification(data.y, params.nrows, params.nclasses, stream);
    }
    // create model
    ML::RandomForestRegressorF rf_model;
    auto* mPtr         = &rf_model;
    mPtr->trees        = nullptr;
    size_t train_nrows = std::min(params.nrows, 1000);
    fit(*handle, mPtr, data.X, train_nrows, params.ncols, data.y, p_rest.rf);
    CUDA_CHECK(hipStreamSynchronize(stream));

    ML::build_treelite_forest(&model, &rf_model, params.ncols, params.nclasses > 1 ? 2 : 1);
    for (int threads_per_tree : {64, 128, 256}) {
      for (int blocks_per_sm : {8}) {
        for (int i : {1, 2}) {
          char *ostr = nullptr;
          bool once = threads_per_tree == 64 && i == 1;
          ML::fil::treelite_params_t tl_params = {
            .algo = p_rest.algo,
            .output_class = params.nclasses > 1,  // cuML RF forest
            .threshold = 1.f / params.nclasses,   //Fixture::DatasetParams
            .storage_type = p_rest.storage,
            .blocks_per_sm = blocks_per_sm,
            .threads_per_tree = threads_per_tree,
            .n_items = 1,
            .pforest_shape_str = once ? &ostr : nullptr};
          ML::fil::from_treelite(*handle, &forest, model, &tl_params);
          if(once) {
            std::cout << ostr << std::endl;
            ::free(ostr);
          }

          hipEvent_t start;
          hipEvent_t stop;
          CUDA_CHECK(hipEventCreate(&start));
          CUDA_CHECK(hipEventCreate(&stop));
          for (int i = 1; i < 10 * p_rest.predict_repetitions; ++i) {
            ML::fil::predict(*this->handle, this->forest, this->data.y,
                             this->data.X, this->params.nrows, false);
          }
          CUDA_CHECK(hipEventRecord(start, 0));
          for (int i = 0; i < p_rest.predict_repetitions; i++) {
            ML::fil::predict(*this->handle, this->forest, this->data.y,
                             this->data.X, this->params.nrows, false);
          }
          CUDA_CHECK_NO_THROW(hipEventRecord(stop, 0));
          CUDA_CHECK_NO_THROW(hipEventSynchronize(stop));
          float milliseconds = 0.0f;
          CUDA_CHECK_NO_THROW(hipEventElapsedTime(&milliseconds, start, stop));
          printf(
            "max_depth %d n_trees %d blocks_per_sm %d threads_per_tree %d %7s "
            "%.2f ms\n",
            p_rest.rf.tree_params.max_depth, p_rest.rf.n_trees, blocks_per_sm,
            threads_per_tree,
            tl_params.storage_type == ML::fil::SPARSE ? "SPARSE16" : "sparse8",
            milliseconds / p_rest.predict_repetitions);
          CUDA_CHECK_NO_THROW(hipEventDestroy(start));
          CUDA_CHECK_NO_THROW(hipEventDestroy(stop));
        }
      }
    }

    // only time prediction
    this->loopOnState(state, [this]() {
      // Dataset<D, L> allocates y assuming one output value per input row,
      // so not supporting predict_proba yet
      for (int i = 0; i < p_rest.predict_repetitions; i++) {
        ML::fil::predict(
          *this->handle, this->forest, this->data.y, this->data.X, this->params.nrows, false);
      }
    });
  }

  void allocateBuffers(const ::benchmark::State& state) override { Base::allocateBuffers(state); }

  void deallocateBuffers(const ::benchmark::State& state) override
  {
    ML::fil::free(*handle, forest);
    Base::deallocateBuffers(state);
  }

 private:
  ML::fil::forest_t forest;
  ModelHandle model;
  Params p_rest;
};

struct FilBenchParams {
  int nrows;
  int ncols;
  int nclasses;
  int max_depth;
  int ntrees;
  ML::fil::storage_type_t storage;
  ML::fil::algo_t algo;
};

std::vector<Params> getInputs()
{
  std::vector<Params> out;
  Params p;
  p.data.rowMajor = true;
  p.blobs         = {.n_informative  = -1,  // Just a placeholder value, anyway changed below
             .effective_rank = -1,  // Just a placeholder value, anyway changed below
             .bias           = 0.f,
             .tail_strength  = 0.1,
             .noise          = 0.01,
             .shuffle        = false,
             .seed           = 12345ULL};

  p.rf = set_rf_params(4,                 /*max_depth */
                       1024,          /* max_leaves */
                       1.f,                /* max_features */
                       32,                 /* n_bins */
                       3,                  /* min_samples_leaf */
                       3,                  /* min_samples_split */
                       0.0f,               /* min_impurity_decrease */
                       true,               /* bootstrap */
                       1,                  /* n_trees */
                       1.f,                /* max_samples */
                       1234ULL,            /* seed */
                       ML::CRITERION::MSE, /* split_criterion */
                       8,                  /* n_streams */
                       128                 /* max_batch_size */
  );

  using ML::fil::algo_t;
  using ML::fil::storage_type_t;
  std::vector<FilBenchParams> var_params = {
    {(int)1e6, 20, 1, 4, 1000, storage_type_t::DENSE, algo_t::TREE_REORG}};
  for (auto& i : var_params) {
    p.data.nrows               = i.nrows;
    p.data.ncols               = i.ncols;
    p.blobs.n_informative      = i.ncols / 3;
    p.blobs.effective_rank     = i.ncols / 3;
    p.data.nclasses            = i.nclasses;
    p.rf.tree_params.max_depth = i.max_depth;
    p.rf.n_trees               = i.ntrees;
    p.storage                  = i.storage;
    p.algo                     = i.algo;
    p.predict_repetitions      = 10;
    out.push_back(p);
  }
  return out;
}

ML_BENCH_REGISTER(Params, FIL, "", getInputs());

}  // end namespace fil
}  // end namespace Bench
}  // end namespace ML
