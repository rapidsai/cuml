#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cmath>
#include <cuml/cuml.hpp>
#include <cuml/ensemble/randomforest.hpp>
#include <utility>
#include "benchmark.cuh"

namespace ML {
namespace Bench {
namespace rf {

struct Params {
  DatasetParams data;
  BlobsParams blobs;
  RF_params rf;
};

template <typename D>
struct RFClassifierModel {};

template <>
struct RFClassifierModel<float> {
  ML::RandomForestClassifierF model;
};

template <>
struct RFClassifierModel<double> {
  ML::RandomForestClassifierD model;
};

template <typename D>
class RFClassifier : public BlobsFixture<D> {
 public:
  RFClassifier(const std::string& name, const Params& p)
    : BlobsFixture<D>(name, p.data, p.blobs), rfParams(p.rf) {}

 protected:
  void runBenchmark(::benchmark::State& state) override {
    using MLCommon::Bench::CudaEventTimer;
    if (this->params.rowMajor) {
      state.SkipWithError("RFClassifier only supports col-major inputs");
    }
    this->loopOnState(state, [this]() {
      auto* mPtr = &model.model;
      mPtr->trees = nullptr;
      fit(*this->handle, mPtr, this->data.X, this->params.nrows,
          this->params.ncols, this->data.y, this->params.nclasses, rfParams);
      CUDA_CHECK(hipStreamSynchronize(this->stream));
    });
  }

 private:
  RFClassifierModel<D> model;
  RF_params rfParams;
};

template <typename D>
std::vector<Params> getInputs() {
  struct Triplets {
    int nrows, ncols, nclasses;
  };
  std::vector<Params> out;
  Params p;
  p.data.rowMajor = false;
  p.blobs = {10.0,         // cluster_std
             false,        // shuffle
             -10.0,        // center_box_min
             10.0,         // center_box_max
             2152953ULL};  //seed

  p.rf = set_rf_params(10,                  /*max_depth */
                       (1 << 20),           /* max_leaves */
                       0.3,                 /* max_features */
                       32,                  /* n_bins */
                       1,                   /* split_algo */
                       3,                   /* min_samples_leaf */
                       3,                   /* min_samples_split */
                       0.0f,                /* min_impurity_decrease */
                       true,                /* bootstrap_features */
                       true,                /* bootstrap */
                       500,                 /* n_trees */
                       1.f,                 /* max_samples */
                       1234ULL,             /* seed */
                       ML::CRITERION::GINI, /* split_criterion */
                       false,               /* quantile_per_tree */
                       8,                   /* n_streams */
                       false,               /* use_experimental_backend */
                       128                  /* max_batch_size */
  );

  std::vector<Triplets> rowcols = {
    {160000, 64, 2},
    {640000, 64, 8},
    {1184000, 968, 2},  // Mimicking Bosch dataset
  };
  for (auto& rc : rowcols) {
    // Let's run Bosch only for float type
    if (!std::is_same<D, float>::value && rc.ncols == 968) continue;
    p.data.nrows = rc.nrows;
    p.data.ncols = rc.ncols;
    p.data.nclasses = rc.nclasses;
    p.rf.tree_params.max_features = 1.f / std::sqrt(float(rc.ncols));
    for (auto max_depth : std::vector<int>({7, 9})) {
      p.rf.tree_params.max_depth = max_depth;
      out.push_back(p);
    }
  }
  return out;
}

ML_BENCH_REGISTER(Params, RFClassifier<float>, "blobs", getInputs<float>());
ML_BENCH_REGISTER(Params, RFClassifier<double>, "blobs", getInputs<double>());

}  // end namespace rf
}  // end namespace Bench
}  // end namespace ML
