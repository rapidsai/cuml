#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/experimental/fil/detail/raft_proto/device_type.hpp>
#include <cuml/experimental/fil/infer_kind.hpp>
#include <cuml/experimental/fil/tree_layout.hpp>
#include <cuml/experimental/fil/treelite_importer.hpp>
#include <cuml/fil/fil.h>

#include "benchmark.cuh"
#include <chrono>
#include <cstdint>
#include <cuml/common/logger.hpp>
#include <cuml/ensemble/randomforest.hpp>
#include <cuml/tree/algo_helper.h>
#include <treelite/c_api.h>
#include <treelite/tree.h>
#include <utility>

namespace ML {
namespace Bench {
namespace filex {

struct Params {
  DatasetParams data;
  RegressionParams blobs;
  ModelHandle model;
  ML::fil::storage_type_t storage;
  bool use_experimental;
  RF_params rf;
  int predict_repetitions;
};

class FILEX : public RegressionFixture<float> {
  typedef RegressionFixture<float> Base;

 public:
  FILEX(const std::string& name, const Params& p)
    : RegressionFixture<float>(name, p.data, p.blobs), model(p.model), p_rest(p)
  {
  }

 protected:
  void runBenchmark(::benchmark::State& state) override
  {
    if (!params.rowMajor) { state.SkipWithError("FIL only supports row-major inputs"); }
    // create model
    ML::RandomForestRegressorF rf_model;
    auto* mPtr       = &rf_model;
    auto train_nrows = std::min(params.nrows, 1000);
    fit(*handle, mPtr, data.X.data(), train_nrows, params.ncols, data.y.data(), p_rest.rf);
    handle->sync_stream(stream);

    ML::build_treelite_forest(&model, &rf_model, params.ncols);

    auto filex_model = ML::experimental::fil::import_from_treelite_handle(
      model,
      ML::experimental::fil::tree_layout::breadth_first,
      128,
      false,
      raft_proto::device_type::gpu,
      0,
      stream);

    ML::fil::treelite_params_t tl_params = {
      .algo              = ML::fil::algo_t::NAIVE,
      .output_class      = false,
      .threshold         = 1.f / params.nclasses,  // Fixture::DatasetParams
      .storage_type      = p_rest.storage,
      .blocks_per_sm     = 8,
      .threads_per_tree  = 1,
      .n_items           = 0,
      .pforest_shape_str = nullptr};
    ML::fil::forest_variant forest_variant;
    auto optimal_chunk_size    = 1;
    auto optimal_storage_type  = p_rest.storage;
    auto optimal_algo_type     = ML::fil::algo_t::NAIVE;
    auto optimal_layout        = ML::experimental::fil::tree_layout::breadth_first;
    auto allowed_storage_types = std::vector<ML::fil::storage_type_t>{};
    if (p_rest.storage == ML::fil::storage_type_t::DENSE) {
      allowed_storage_types.push_back(ML::fil::storage_type_t::DENSE);
      allowed_storage_types.push_back(ML::fil::storage_type_t::SPARSE);
      allowed_storage_types.push_back(ML::fil::storage_type_t::SPARSE8);
    } else {
      allowed_storage_types.push_back(ML::fil::storage_type_t::SPARSE);
      allowed_storage_types.push_back(ML::fil::storage_type_t::SPARSE8);
    }
    auto allowed_layouts = std::vector<ML::experimental::fil::tree_layout>{
      ML::experimental::fil::tree_layout::breadth_first,
      ML::experimental::fil::tree_layout::depth_first,
    };
    auto min_time = std::numeric_limits<std::int64_t>::max();

    // Iterate through storage type, algorithm type, and chunk sizes and find optimum
    for (auto storage_type : allowed_storage_types) {
      auto allowed_algo_types = std::vector<ML::fil::algo_t>{};
      allowed_algo_types.push_back(ML::fil::algo_t::NAIVE);
      if (storage_type == ML::fil::storage_type_t::DENSE) {
        allowed_algo_types.push_back(ML::fil::algo_t::TREE_REORG);
        allowed_algo_types.push_back(ML::fil::algo_t::BATCH_TREE_REORG);
      }
      tl_params.storage_type = storage_type;

      for (auto algo_type : allowed_algo_types) {
        tl_params.algo = algo_type;
        for (auto layout : allowed_layouts) {
          filex_model = ML::experimental::fil::import_from_treelite_handle(
            model, layout, 128, false, raft_proto::device_type::gpu, 0, stream);
          for (auto chunk_size = 1; chunk_size <= 32; chunk_size *= 2) {
            if (!p_rest.use_experimental) {
              tl_params.threads_per_tree = chunk_size;
              ML::fil::from_treelite(*handle, &forest_variant, model, &tl_params);
              forest = std::get<ML::fil::forest_t<float>>(forest_variant);
            }
            handle->sync_stream();
            handle->sync_stream_pool();
            auto start = std::chrono::high_resolution_clock::now();
            for (int i = 0; i < p_rest.predict_repetitions; i++) {
              // Create FIL forest
              if (p_rest.use_experimental) {
                filex_model.predict(*handle,
                                    data.y.data(),
                                    data.X.data(),
                                    params.nrows,
                                    raft_proto::device_type::gpu,
                                    raft_proto::device_type::gpu,
                                    ML::experimental::fil::infer_kind::default_kind,
                                    chunk_size);
              } else {
                ML::fil::predict(
                  *handle, forest, data.y.data(), data.X.data(), params.nrows, false);
              }
            }
            handle->sync_stream();
            handle->sync_stream_pool();
            auto end = std::chrono::high_resolution_clock::now();
            auto elapsed =
              std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
            if (elapsed < min_time) {
              min_time             = elapsed;
              optimal_chunk_size   = chunk_size;
              optimal_storage_type = storage_type;
              optimal_algo_type    = algo_type;
              optimal_layout       = layout;
            }

            // Clean up from FIL
            if (!p_rest.use_experimental) { ML::fil::free(*handle, forest); }
          }
          if (!p_rest.use_experimental) { break; }
        }
        if (p_rest.use_experimental) { break; }
      }
      if (p_rest.use_experimental) { break; }
    }

    // Build optimal FIL tree
    tl_params.storage_type     = optimal_storage_type;
    tl_params.algo             = optimal_algo_type;
    tl_params.threads_per_tree = optimal_chunk_size;
    ML::fil::from_treelite(*handle, &forest_variant, model, &tl_params);
    forest      = std::get<ML::fil::forest_t<float>>(forest_variant);
    filex_model = ML::experimental::fil::import_from_treelite_handle(
      model, optimal_layout, 128, false, raft_proto::device_type::gpu, 0, stream);

    handle->sync_stream();
    handle->sync_stream_pool();

    // only time prediction
    this->loopOnState(
      state,
      [this, &filex_model, optimal_chunk_size]() {
        for (int i = 0; i < p_rest.predict_repetitions; i++) {
          if (p_rest.use_experimental) {
            filex_model.predict(*handle,
                                this->data.y.data(),
                                this->data.X.data(),
                                this->params.nrows,
                                raft_proto::device_type::gpu,
                                raft_proto::device_type::gpu,
                                ML::experimental::fil::infer_kind::default_kind,
                                optimal_chunk_size);
            handle->sync_stream();
            handle->sync_stream_pool();
          } else {
            ML::fil::predict(*this->handle,
                             this->forest,
                             this->data.y.data(),
                             this->data.X.data(),
                             this->params.nrows,
                             false);
            handle->sync_stream();
            handle->sync_stream_pool();
          }
        }
      },
      true);
  }

  void allocateBuffers(const ::benchmark::State& state) override { Base::allocateBuffers(state); }

  void deallocateBuffers(const ::benchmark::State& state) override
  {
    ML::fil::free(*handle, forest);
    Base::deallocateBuffers(state);
  }

 private:
  ML::fil::forest_t<float> forest;
  ModelHandle model;
  Params p_rest;
};

struct FilBenchParams {
  int nrows;
  int ncols;
  int nclasses;
  int max_depth;
  int ntrees;
  ML::fil::storage_type_t storage;
  bool use_experimental;
};

std::vector<Params> getInputs()
{
  std::vector<Params> out;
  Params p;
  p.data.rowMajor = true;
  p.blobs         = {.n_informative  = -1,  // Just a placeholder value, anyway changed below
                     .effective_rank = -1,  // Just a placeholder value, anyway changed below
                     .bias           = 0.f,
                     .tail_strength  = 0.1,
                     .noise          = 0.01,
                     .shuffle        = false,
                     .seed           = 12345ULL};

  p.rf = set_rf_params(10,                 /*max_depth */
                       (1 << 20),          /* max_leaves */
                       1.f,                /* max_features */
                       32,                 /* max_n_bins */
                       3,                  /* min_samples_leaf */
                       3,                  /* min_samples_split */
                       0.0f,               /* min_impurity_decrease */
                       true,               /* bootstrap */
                       1,                  /* n_trees */
                       1.f,                /* max_samples */
                       1234ULL,            /* seed */
                       ML::CRITERION::MSE, /* split_criterion */
                       8,                  /* n_streams */
                       128                 /* max_batch_size */
  );

  using ML::fil::algo_t;
  using ML::fil::storage_type_t;
  std::vector<FilBenchParams> var_params = {
    {(int)1e6, 20, 1, 10, 1000, storage_type_t::DENSE, false},
    {(int)1e6, 20, 1, 10, 1000, storage_type_t::DENSE, true},
    {(int)1e6, 20, 1, 3, 1000, storage_type_t::DENSE, false},
    {(int)1e6, 20, 1, 3, 1000, storage_type_t::DENSE, true},
    {(int)1e6, 20, 1, 28, 1000, storage_type_t::SPARSE, false},
    {(int)1e6, 20, 1, 28, 1000, storage_type_t::SPARSE, true},
    {(int)1e6, 20, 1, 10, 100, storage_type_t::DENSE, false},
    {(int)1e6, 20, 1, 10, 100, storage_type_t::DENSE, true},
    {(int)1e6, 20, 1, 10, 10000, storage_type_t::DENSE, false},
    {(int)1e6, 20, 1, 10, 10000, storage_type_t::DENSE, true},
    {(int)1e6, 200, 1, 10, 1000, storage_type_t::DENSE, false},
    {(int)1e6, 200, 1, 10, 1000, storage_type_t::DENSE, true}};
  for (auto& i : var_params) {
    p.data.nrows               = i.nrows;
    p.data.ncols               = i.ncols;
    p.blobs.n_informative      = i.ncols / 3;
    p.blobs.effective_rank     = i.ncols / 3;
    p.data.nclasses            = i.nclasses;
    p.rf.tree_params.max_depth = i.max_depth;
    p.rf.n_trees               = i.ntrees;
    p.storage                  = i.storage;
    p.use_experimental         = i.use_experimental;
    p.predict_repetitions      = 10;
    out.push_back(p);
  }
  return out;
}

ML_BENCH_REGISTER(Params, FILEX, "", getInputs());

}  // namespace filex
}  // end namespace Bench
}  // end namespace ML
