#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/ml_benchmark.hpp>
#include <cuml/matrix/kernelparams.h>
#include <matrix/grammatrix.cuh>
#include <matrix/kernelfactory.cuh>
#include <memory>
// #TODO: Replace with public header when ready
#include <raft/linalg/detail/cublas_wrappers.hpp>
#include <raft/random/rng.hpp>
#include <sstream>
#include <string>
#include <vector>

namespace MLCommon {
namespace Bench {
namespace Matrix {

using namespace MLCommon::Matrix;

struct GramTestParams {
  int m;  // m parameter of the GEMM
  int k;  // k parameter of the GEMM
  int n;  // n parameter of the GEMM
  KernelParams kernel_params;
  bool is_row_major;
};  // struct GramTestParams

template <typename T>
struct GramMatrix : public Fixture {
  GramMatrix(const std::string& name, const GramTestParams& p)
    : Fixture(name), params(p), A(0, stream), B(0, stream), C(0, stream)
  {
    std::vector<std::string> kernel_names{"linear", "poly", "rbf", "tanh"};
    std::ostringstream oss;
    oss << name << "/" << kernel_names[p.kernel_params.kernel] << "/" << p.m << "x" << p.k << "x"
        << p.n << "/" << (p.is_row_major ? "row_major" : "col_major");
    this->SetName(oss.str().c_str());

    RAFT_CUBLAS_TRY(hipblasCreate(&cublas_handle));
    kernel =
      std::unique_ptr<GramMatrixBase<T>>(KernelFactory<T>::create(p.kernel_params, cublas_handle));
  }

  ~GramMatrix() { RAFT_CUBLAS_TRY_NO_THROW(hipblasDestroy(cublas_handle)); }

 protected:
  void allocateBuffers(const ::benchmark::State& state) override
  {
    A.resize(params.m * params.k, stream);
    B.resize(params.k * params.n, stream);
    C.resize(params.m * params.n, stream);
    raft::random::Rng r(123456ULL);
    r.uniform(A.data(), params.m * params.k, T(-1.0), T(1.0), stream);
    r.uniform(B.data(), params.k * params.n, T(-1.0), T(1.0), stream);
  }
  void deallocateBuffers(const ::benchmark::State& state) override
  {
    A.release();
    B.release();
    C.release();
  }
  void runBenchmark(::benchmark::State& state) override
  {
    if (!this->kernel) { state.SkipWithError("Kernel matrix is not initialized"); }
    loopOnState(state, [this]() {
      (*this->kernel)(A.data(),
                      this->params.m,
                      this->params.k,
                      B.data(),
                      this->params.n,
                      C.data(),
                      this->params.is_row_major,
                      this->stream);
    });
  }

 private:
  hipblasHandle_t cublas_handle;
  std::unique_ptr<GramMatrixBase<T>> kernel;
  GramTestParams params;

  rmm::device_uvector<T> A;  // input matrix A, size [m * k]
  rmm::device_uvector<T> B;  // input matrix B, size [n * k]
  rmm::device_uvector<T> C;  // output matrix C, size [m*n]
};

static std::vector<GramTestParams> getInputs()
{
  std::vector<GramTestParams> param_vec;
  std::vector<KernelParams> kernel_params{KernelParams{LINEAR, 3, 1, 0},
                                          KernelParams{POLYNOMIAL, 2, 1.3, 1},
                                          KernelParams{TANH, 2, 0.5, 2.4},
                                          KernelParams{RBF, 2, 0.5, 0}};
  struct TestSize {
    int m;
    int k;
    int n;
  };
  std::vector<TestSize> data_size{{4096, 10, 1024},
                                  {4096, 100, 1024},
                                  {4096, 1000, 1024},
                                  {4096, 10000, 1024},
                                  {100000, 10, 1024},
                                  {100000, 100, 1024},
                                  {100000, 1000, 1024}};

  param_vec.reserve(kernel_params.size() * data_size.size());
  for (TestSize s : data_size) {
    for (auto kernel : kernel_params) {
      for (bool row_major : {false, true}) {
        param_vec.push_back(GramTestParams{s.m, s.k, s.n, kernel, row_major});
      }
    }
  }
  return param_vec;
}

ML_BENCH_REGISTER(GramTestParams, GramMatrix<float>, "", getInputs());
ML_BENCH_REGISTER(GramTestParams, GramMatrix<double>, "", getInputs());

}  // namespace Matrix
}  // namespace Bench
}  // namespace MLCommon
