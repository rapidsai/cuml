#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuml/matrix/kernelparams.h>
#include <matrix/grammatrix.cuh>
#include <matrix/kernelfactory.cuh>
#include <memory>
#include <raft/random/rng.cuh>
#include <sstream>
#include <string>
#include <vector>
#include "../common/ml_benchmark.hpp"

namespace MLCommon {
namespace Bench {
namespace Matrix {

using namespace MLCommon::Matrix;

struct GramTestParams {
  int m;  // m parameter of the GEMM
  int k;  // k parameter of the GEMM
  int n;  // n parameter of the GEMM
  KernelParams kernel_params;
};  // struct GramTestParams

template <typename T>
struct GramMatrix : public Fixture {
  GramMatrix(const std::string& name, const GramTestParams& p)
    : Fixture(name, std::shared_ptr<deviceAllocator>(
                      new raft::mr::device::default_allocator)),
      params(p) {
    std::vector<std::string> kernel_names{"linear", "poly", "rbf", "tanh"};
    std::ostringstream oss;
    oss << name << "/" << kernel_names[p.kernel_params.kernel] << "/" << p.m
        << "x" << p.k << "x" << p.n;
    this->SetName(oss.str().c_str());

    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    kernel = std::unique_ptr<GramMatrixBase<T>>(
      KernelFactory<T>::create(p.kernel_params, cublas_handle));
  }

  ~GramMatrix() { CUBLAS_CHECK(hipblasDestroy(cublas_handle)); }

 protected:
  void allocateBuffers(const ::benchmark::State& state) override {
    alloc(A, params.m * params.k);
    alloc(B, params.k * params.n);
    alloc(C, params.m * params.n);
    raft::random::Rng r(123456ULL);
    r.uniform(A, params.m * params.k, T(-1.0), T(1.0), stream);
    r.uniform(B, params.k * params.n, T(-1.0), T(1.0), stream);
  }

  void deallocateBuffers(const ::benchmark::State& state) override {
    dealloc(A, params.m * params.k);
    dealloc(B, params.k * params.n);
    dealloc(C, params.m * params.n);
  }

  void runBenchmark(::benchmark::State& state) override {
    if (!this->kernel) {
      state.SkipWithError("Kernel matrix is not initialized");
    }
    loopOnState(state, [this]() {
      (*this->kernel)(this->A, this->params.m, this->params.k, this->B,
                      this->params.n, this->C, this->stream);
    });
  }

 private:
  hipblasHandle_t cublas_handle;
  std::unique_ptr<GramMatrixBase<T>> kernel;
  GramTestParams params;

  T* A;  // input matrix A, size [m * k]
  T* B;  // input matrix B, size [n * k]
  T* C;  // output matrix C, size [m*n]
};

static std::vector<GramTestParams> getInputs() {
  std::vector<GramTestParams> param_vec;
  std::vector<KernelParams> kernel_params{
    KernelParams{LINEAR, 3, 1, 0}, KernelParams{POLYNOMIAL, 2, 1.3, 1},
    KernelParams{TANH, 2, 0.5, 2.4}, KernelParams{RBF, 2, 0.5, 0}};
  struct TestSize {
    int m;
    int k;
    int n;
  };
  std::vector<TestSize> data_size{{4096, 10, 1024},    {4096, 100, 1024},
                                  {4096, 1000, 1024},  {4096, 10000, 1024},
                                  {100000, 10, 1024},  {100000, 100, 1024},
                                  {100000, 1000, 1024}};

  param_vec.reserve(kernel_params.size() * data_size.size());
  for (TestSize s : data_size) {
    for (auto kernel : kernel_params) {
      param_vec.push_back(GramTestParams{s.m, s.k, s.n, kernel});
    }
  }
  return param_vec;
}

ML_BENCH_REGISTER(GramTestParams, GramMatrix<float>, "", getInputs());
ML_BENCH_REGISTER(GramTestParams, GramMatrix<double>, "", getInputs());

}  // namespace Matrix
}  // namespace Bench
}  // namespace MLCommon
