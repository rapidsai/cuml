/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <random/permute.h>
#include <random/rng.h>
#include "benchmark.cuh"

namespace MLCommon {
namespace Bench {
namespace Random {

struct Params {
  int rows, cols;
  bool needPerms, needShuffle, rowMajor;
};  // struct Params

template <typename T>
struct Permute : public Fixture {
  Permute(const std::string& name, const Params& p)
    : Fixture(name), params(p) {}

 protected:
  void allocateBuffers(const ::benchmark::State& state) override {
    auto matLen = params.rows * params.cols;
    auto vecLen = params.rows;
    if (params.needPerms) {
      allocate(perms, vecLen);
    } else {
      perms = nullptr;
    }
    MLCommon::Random::Rng r(123456ULL);
    if (params.needShuffle) {
      allocate(out, matLen);
      allocate(in, matLen);
      r.uniform(in, vecLen, T(-1.0), T(1.0), stream);
    } else {
      out = in = nullptr;
    }
  }

  void deallocateBuffers(const ::benchmark::State& state) override {
    if (params.needShuffle) {
      CUDA_CHECK(hipFree(out));
      CUDA_CHECK(hipFree(in));
    }
    if (params.needPerms) {
      CUDA_CHECK(hipFree(perms));
    }
  }

  void runBenchmark(::benchmark::State& state) override {
    MLCommon::Random::Rng r(123456ULL);
    for (auto _ : state) {
      CudaEventTimer timer(state, scratchBuffer, stream);
      MLCommon::Random::permute(perms, out, in, params.cols, params.rows,
                                params.rowMajor, stream);
    }
  }

 private:
  Params params;
  T *out, *in;
  int* perms;
};  // struct Permute

static std::vector<Params> getInputs() {
  return {
    {32 * 1024, 128, true, true, true},
    {1024 * 1024, 128, true, true, true},
    {32 * 1024, 128 + 2, true, true, true},
    {1024 * 1024, 128 + 2, true, true, true},
    {32 * 1024, 128 + 1, true, true, true},
    {1024 * 1024, 128 + 1, true, true, true},

    {32 * 1024, 128, true, true, false},
    {1024 * 1024, 128, true, true, false},
    {32 * 1024, 128 + 2, true, true, false},
    {1024 * 1024, 128 + 2, true, true, false},
    {32 * 1024, 128 + 1, true, true, false},
    {1024 * 1024, 128 + 1, true, true, false},
  };
}

PRIMS_BENCH_REGISTER(Params, Permute<float>, "permute", getInputs());
PRIMS_BENCH_REGISTER(Params, Permute<double>, "permute", getInputs());

}  // namespace Random
}  // namespace Bench
}  // namespace MLCommon
