#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/cudart_utils.h>
#include <limits>
#include <raft/distance/fused_l2_nn.cuh>
#include <raft/linalg/norm.cuh>
#include <raft/random/rng.cuh>
#include "../common/ml_benchmark.hpp"

namespace MLCommon {
namespace Bench {
namespace Distance {

struct FLNParams {
  int m, n, k;
};  // struct FLNParams

template <typename T>
struct FusedL2NN : public Fixture {
  FusedL2NN(const std::string& name, const FLNParams& p)
    : Fixture(name, std::shared_ptr<deviceAllocator>(
                      new raft::mr::device::default_allocator)),
      params(p) {}

 protected:
  void allocateBuffers(const ::benchmark::State& state) override {
    alloc(x, params.m * params.k);
    alloc(y, params.n * params.k);
    alloc(xn, params.m);
    alloc(yn, params.n);
    alloc(out, params.m);
    alloc(workspace, params.m);
    raft::random::Rng r(123456ULL);
    r.uniform(x, params.m * params.k, T(-1.0), T(1.0), stream);
    r.uniform(y, params.n * params.k, T(-1.0), T(1.0), stream);
    raft::linalg::rowNorm(xn, x, params.k, params.m, raft::linalg::L2Norm, true,
                          stream);
    raft::linalg::rowNorm(yn, y, params.k, params.n, raft::linalg::L2Norm, true,
                          stream);
    auto blks = raft::ceildiv(params.m, 256);
    raft::distance::initKernel<T, hipcub::KeyValuePair<int, T>, int>
      <<<blks, 256, 0, stream>>>(out, params.m, std::numeric_limits<T>::max(),
                                 op);
  }

  void deallocateBuffers(const ::benchmark::State& state) override {
    dealloc(x, params.m * params.k);
    dealloc(y, params.n * params.k);
    dealloc(xn, params.m);
    dealloc(yn, params.n);
    dealloc(out, params.m);
    dealloc(workspace, params.m);
  }

  void runBenchmark(::benchmark::State& state) override {
    loopOnState(state, [this]() {
      // it is enough to only benchmark the L2-squared metric
      raft::distance::fusedL2NN<T, hipcub::KeyValuePair<int, T>, int>(
        out, x, y, xn, yn, params.m, params.n, params.k, (void*)workspace, op,
        pairRedOp, false, false, stream);
    });
  }

 private:
  FLNParams params;
  T *x, *y, *xn, *yn;
  hipcub::KeyValuePair<int, T>* out;
  int* workspace;
  raft::distance::KVPMinReduce<int, T> pairRedOp;
  raft::distance::MinAndDistanceReduceOp<int, T> op;
};  // struct FusedL2NN

static std::vector<FLNParams> getInputs() {
  return {
    {32, 16384, 16384},    {64, 16384, 16384},  {128, 16384, 16384},
    {256, 16384, 16384},   {512, 16384, 16384}, {1024, 16384, 16384},
    {16384, 32, 16384},    {16384, 64, 16384},  {16384, 128, 16384},
    {16384, 256, 16384},   {16384, 512, 16384}, {16384, 1024, 16384},
    {16384, 16384, 32},    {16384, 16384, 64},  {16384, 16384, 128},
    {16384, 16384, 256},   {16384, 16384, 512}, {16384, 16384, 1024},
    {16384, 16384, 16384},
  };
}

ML_BENCH_REGISTER(FLNParams, FusedL2NN<float>, "", getInputs());
ML_BENCH_REGISTER(FLNParams, FusedL2NN<double>, "", getInputs());

}  // namespace Distance
}  // namespace Bench
}  // namespace MLCommon
