/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/cudart_utils.h>
#include <raft/random/rng.cuh>
#include "../common/ml_benchmark.hpp"

namespace MLCommon {
namespace Bench {
namespace Random {

enum RandomType {
  RNG_Normal,
  RNG_LogNormal,
  RNG_Uniform,
  RNG_Gumbel,
  RNG_Logistic,
  RNG_Exp,
  RNG_Rayleigh,
  RNG_Laplace,
  RNG_Fill
};  // enum RandomType

template <typename T>
struct Params {
  int len;
  RandomType type;
  raft::random::GeneratorType gtype;
  T start, end;
};  // struct Params

template <typename T>
struct RngBench : public Fixture {
  RngBench(const std::string& name, const Params<T>& p)
    : Fixture(name, std::shared_ptr<deviceAllocator>(
                      new raft::mr::device::default_allocator)),
      params(p) {}

 protected:
  void allocateBuffers(const ::benchmark::State& state) override {
    alloc(ptr, params.len);
  }

  void deallocateBuffers(const ::benchmark::State& state) override {
    dealloc(ptr, params.len);
  }

  void runBenchmark(::benchmark::State& state) override {
    raft::random::Rng r(123456ULL, params.gtype);
    loopOnState(state, [this, &r]() {
      switch (params.type) {
        case RNG_Normal:
          r.normal(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_LogNormal:
          r.lognormal(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Uniform:
          r.uniform(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Gumbel:
          r.gumbel(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Logistic:
          r.logistic(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Exp:
          r.exponential(ptr, params.len, params.start, stream);
          break;
        case RNG_Rayleigh:
          r.rayleigh(ptr, params.len, params.start, stream);
          break;
        case RNG_Laplace:
          r.laplace(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Fill:
          r.fill(ptr, params.len, params.start, stream);
          break;
      };
    });
  }

 private:
  Params<T> params;
  T* ptr;
};  // struct RngBench

template <typename T>
static std::vector<Params<T>> getInputs() {
  using namespace raft::random;
  return {
    {1024 * 1024, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},

    {1024 * 1024, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},

    {1024 * 1024, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},

    {1024 * 1024, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
  };
}

ML_BENCH_REGISTER(Params<float>, RngBench<float>, "", getInputs<float>());
ML_BENCH_REGISTER(Params<double>, RngBench<double>, "", getInputs<double>());

}  // namespace Random
}  // namespace Bench
}  // namespace MLCommon
