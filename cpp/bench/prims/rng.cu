/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <random/rng.h>
#include "benchmark.cuh"

namespace MLCommon {
namespace Bench {
namespace Random {

enum RandomType {
  RNG_Normal,
  RNG_LogNormal,
  RNG_Uniform,
  RNG_Gumbel,
  RNG_Logistic,
  RNG_Exp,
  RNG_Rayleigh,
  RNG_Laplace,
  RNG_Fill
};  // enum RandomType

template <typename T>
struct Params {
  int len;
  RandomType type;
  MLCommon::Random::GeneratorType gtype;
  T start, end;
};  // struct Params

template <typename T>
struct RngBench : public Fixture {
  RngBench(const std::string& name, const Params<T>& p)
    : Fixture(name), params(p) {}

 protected:
  void allocateBuffers(const ::benchmark::State& state) override {
    allocate(ptr, params.len);
  }

  void deallocateBuffers(const ::benchmark::State& state) override {
    CUDA_CHECK(hipFree(ptr));
  }

  void runBenchmark(::benchmark::State& state) override {
    MLCommon::Random::Rng r(123456ULL, params.gtype);
    for (auto _ : state) {
      CudaEventTimer timer(state, scratchBuffer, stream);
      switch (params.type) {
        case RNG_Normal:
          r.normal(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_LogNormal:
          r.lognormal(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Uniform:
          r.uniform(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Gumbel:
          r.gumbel(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Logistic:
          r.logistic(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Exp:
          r.exponential(ptr, params.len, params.start, stream);
          break;
        case RNG_Rayleigh:
          r.rayleigh(ptr, params.len, params.start, stream);
          break;
        case RNG_Laplace:
          r.laplace(ptr, params.len, params.start, params.end, stream);
          break;
        case RNG_Fill:
          r.fill(ptr, params.len, params.start, stream);
          break;
      };
    }
  }

 private:
  Params<T> params;
  T* ptr;
};  // struct RngBench

template <typename T>
static std::vector<Params<T>> getInputs() {
  using namespace MLCommon::Random;
  return {
    {1024 * 1024, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Uniform, GenPhilox, T(-1.0), T(1.0)},

    {1024 * 1024, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Uniform, GenTaps, T(-1.0), T(1.0)},

    {1024 * 1024, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Uniform, GenKiss99, T(-1.0), T(1.0)},

    {1024 * 1024, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 2, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 2, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 2, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 + 1, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {32 * 1024 * 1024 + 1, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
    {1024 * 1024 * 1024 + 1, RNG_Fill, GenPhilox, T(-1.0), T(1.0)},
  };
}

PRIMS_BENCH_REGISTER(Params<float>, RngBench<float>, "rng", getInputs<float>());
PRIMS_BENCH_REGISTER(Params<double>, RngBench<double>, "rng",
                     getInputs<double>());

}  // namespace Random
}  // namespace Bench
}  // namespace MLCommon
