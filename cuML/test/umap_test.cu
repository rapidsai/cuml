/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "umap/umapparams.h"
#include "umap/runner.h"
#include "knn/knn.h"

#include <cuda_utils.h>

#include <vector>
#include <iostream>

using namespace ML;
using namespace std;

/**
 * For now, this is mostly to test the c++ algorithm is able to be built.
 * Comprehensive comparisons of resulting embeddings are being done in the
 * Python test suite. Next to come will be a CUDA implementation of t-SNE's
 * trustworthiness score, which will allow us to gtest embedding algorithms.
 */
class UMAPTest: public ::testing::Test {
protected:
	void basicTest() {

		umap_params = new UMAPParams();
		umap_params->n_neighbors = k;

		kNN *knn = new kNN(d);
		hipStream_t stream;
		CUDA_CHECK(hipStreamCreate(&stream));
		UMAPAlgo::find_ab(umap_params, stream);

		std::vector<float> X = {
			1.0, 1.0, 34.0,
			76.0, 2.0, 29.0,
			34.0, 3.0, 13.0,
			23.0, 7.0, 80.0
		};

    float* X_d;

		MLCommon::allocate(X_d, n*d);
		MLCommon::updateDevice(X_d, X.data(), n*d);

		MLCommon::allocate(embeddings, n*umap_params->n_components);

		UMAPAlgo::_fit<float, 256>(X_d, n, d, knn, umap_params, embeddings, stream);

		float *xformed;
		MLCommon::allocate(xformed, n*umap_params->n_components);

		UMAPAlgo::_transform<float, 256>(X_d, n, d, embeddings, n, knn, umap_params, xformed, stream);
		CUDA_CHECK(hipStreamDestroy(stream));
	}

	void SetUp() override {
		basicTest();
	}

	void TearDown() override {
	}

protected:

	UMAPParams *umap_params;

	int d = 3;
	int n = 4;
	int k = 2;

	float *embeddings;

};


typedef UMAPTest UMAPTestF;
TEST_F(UMAPTestF, Result) {}

