#include "hip/hip_runtime.h"
#include "test_utils.h"
#include "utils.h"
#include <glm/glm.hpp>
#include <glm/qn/glm_linear.h>
#include <glm/qn/glm_logistic.h>
#include <glm/qn/glm_softmax.h>
#include <glm/qn/qn.h>
#include <gtest/gtest.h>
#include <linalg/transpose.h>
#include <vector>

namespace ML {
namespace GLM {

using namespace MLCommon;

struct QuasiNewtonTest : ::testing::Test {
  static constexpr int N = 10;
  static constexpr int D = 2;

  const static double *nobptr;
  const static double tol;
  const static double X[N][D];
  cumlHandle cuml_handle;
  const cumlHandle_impl &handle;
  hipStream_t stream;
  std::shared_ptr<SimpleMatOwning<double>> Xdev;
  std::shared_ptr<SimpleVecOwning<double>> ydev;

  std::shared_ptr<deviceAllocator> allocator;
  QuasiNewtonTest() : handle(cuml_handle.getImpl()) {}
  void SetUp() {
    stream = cuml_handle.getStream();
    Xdev.reset(new SimpleMatOwning<double>(handle.getDeviceAllocator(), N, D,
                                           stream, ROW_MAJOR));
    updateDeviceAsync(Xdev->data, &X[0][0], Xdev->len, stream);

    ydev.reset(
        new SimpleVecOwning<double>(handle.getDeviceAllocator(), N, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    allocator = handle.getDeviceAllocator();
  }
  void TearDown() {}
};

const double *QuasiNewtonTest::nobptr = 0;
const double QuasiNewtonTest::tol = 5e-6;
const double QuasiNewtonTest::X[QuasiNewtonTest::N][QuasiNewtonTest::D] = {
    {-0.2047076594847130, 0.4789433380575482},
    {-0.5194387150567381, -0.5557303043474900},
    {1.9657805725027142, 1.3934058329729904},
    {0.0929078767437177, 0.2817461528302025},
    {0.7690225676118387, 1.2464347363862822},
    {1.0071893575830049, -1.2962211091122635},
    {0.2749916334321240, 0.2289128789353159},
    {1.3529168351654497, 0.8864293405915888},
    {-2.0016373096603974, -0.3718425371402544},
    {1.6690253095248706, -0.4385697358355719}};

template <typename T, class Comp>
::testing::AssertionResult
checkParamsEqual(const cumlHandle_impl &handle, const T *host_weights,
                 const T *host_bias, const T *w, const GLMDims &dims,
                 Comp &comp, hipStream_t stream) {
  int C = dims.C;
  int D = dims.D;
  bool fit_intercept = dims.fit_intercept;
  std::vector<T> w_ref_cm(C * D);
  int idx = 0;
  for (int d = 0; d < D; d++)
    for (int c = 0; c < C; c++) {
      w_ref_cm[idx++] = host_weights[c * D + d];
    }

  SimpleVecOwning<T> w_ref(handle.getDeviceAllocator(), dims.n_param, stream);
  updateDeviceAsync(w_ref.data, &w_ref_cm[0], C * D, stream);
  if (fit_intercept) {
    updateDeviceAsync(&w_ref.data[C * D], host_bias, C, stream);
  }
  CUDA_CHECK(hipStreamSynchronize(stream));
  return devArrMatch(w_ref.data, w, w_ref.len, comp);
}

template <typename T, class LossFunction>
T run(const cumlHandle_impl &handle, LossFunction &loss, const SimpleMat<T> &X,
      const SimpleVec<T> &y, T l1, T l2, T *w, SimpleMat<T> &z, int verbosity,
      hipStream_t stream) {

  int max_iter = 100;
  T grad_tol = 1e-8;
  int linesearch_max_iter = 50;
  int lbfgs_memory = 5;
  int num_iters = 0;

  T fx;
  SimpleVec<T> w0(w, loss.n_param);

  qn_fit<T, LossFunction>(handle, loss, X.data, y.data, z.data, X.m, l1, l2,
                          max_iter, grad_tol, linesearch_max_iter, lbfgs_memory,
                          verbosity, w0.data, &fx, &num_iters, X.ord, stream);

  return fx;
}

template <typename T>
T run_api(const cumlHandle &cuml_handle, int loss_type, int C,
          bool fit_intercept, const SimpleMat<T> &X, const SimpleVec<T> &y,
          T l1, T l2, T *w, SimpleMat<T> &z, int verbosity,
          hipStream_t stream) {

  int max_iter = 100;
  T grad_tol = 1e-8;
  int linesearch_max_iter = 50;
  int lbfgs_memory = 5;
  int num_iters = 0;

  SimpleVec<T> w0(w, X.n + fit_intercept);
  w0.fill(T(0), stream);
  T fx;

  qnFit(cuml_handle, X.data, y.data, X.m, X.n, C, fit_intercept, l1, l2,
        max_iter, grad_tol, linesearch_max_iter, lbfgs_memory, verbosity, w,
        &fx, &num_iters, false, loss_type);

  return fx;
}

TEST_F(QuasiNewtonTest, binary_logistic_vs_sklearn) {
  CompareApprox<double> compApprox(tol);
  // Test case generated in python and solved with sklearn
  double y[N] = {1, 1, 1, 0, 1, 0, 1, 0, 1, 0};
  updateDeviceAsync(ydev->data, &y[0], ydev->len, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  double alpha = 0.01;

  LogisticLoss<double> loss_b(handle, D, true);
  LogisticLoss<double> loss_no_b(handle, D, false);

  SimpleVecOwning<double> w0(allocator, D + 1, stream);
  SimpleVecOwning<double> z(allocator, N, stream);

  double l1, l2, fx;

  double w_l1_b[2] = {-1.6899370396155091, 1.9021577534928300};
  double b_l1_b = 0.8057670813749118;
  double obj_l1_b = 0.44295941481024703;

  l1 = alpha;
  l2 = 0.0;
  fx = run(handle, loss_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l1_b[0], &b_l1_b, w0.data, loss_b,
                               compApprox, stream));

  fx = run_api(cuml_handle, 0, 1, loss_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_b, fx));

  double w_l2_b[2] = {-1.5339880402781370, 1.6788639581350926};
  double b_l2_b = 0.806087868102401;
  double obj_l2_b = 0.4378085369889721;

  l1 = 0;
  l2 = alpha;
  fx = run(handle, loss_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);

  ASSERT_TRUE(compApprox(obj_l2_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l2_b[0], &b_l2_b, w0.data, loss_b,
                               compApprox, stream));

  fx = run_api(cuml_handle, 0, 1, loss_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_b, fx));

  double w_l1_no_b[2] = {-1.6215035298864591, 2.3650868394981086};
  double obj_l1_no_b = 0.4769896009200278;

  l1 = alpha;
  l2 = 0.0;
  fx = run(handle, loss_no_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_no_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l1_no_b[0], nobptr, w0.data,
                               loss_no_b, compApprox, stream));

  fx = run_api(cuml_handle, 0, 1, loss_no_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_no_b, fx));

  double w_l2_no_b[2] = {-1.3931049893764620, 2.0140103094119621};
  double obj_l2_no_b = 0.47502098062114273;

  l1 = 0;
  l2 = alpha;
  fx = run(handle, loss_no_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_no_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l2_no_b[0], nobptr, w0.data,
                               loss_no_b, compApprox, stream));

  fx = run_api(cuml_handle, 0, 1, loss_no_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_no_b, fx));
}

TEST_F(QuasiNewtonTest, multiclass_logistic_vs_sklearn) {
  // The data seems to small for the objective to be strongly convex
  // leaving out exact param checks

  CompareApprox<double> compApprox(tol);
  double y[N] = {2, 2, 0, 3, 3, 0, 0, 0, 1, 0};
  updateDeviceAsync(ydev->data, &y[0], ydev->len, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  double fx, l1, l2;
  int C = 4;

  double alpha = 0.016;

  SimpleMatOwning<double> z(allocator, C, N, stream);
  SimpleVecOwning<double> w0(allocator, C * (D + 1), stream);

  Softmax<double> loss_b(handle, D, C, true);
  Softmax<double> loss_no_b(handle, D, C, false);

  l1 = alpha;
  l2 = 0.0;
  double obj_l1_b = 0.5407911382311313;

  fx = run(handle, loss_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_b, fx));

  fx = run_api(cuml_handle, 2, C, loss_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_b, fx));

  l1 = 0.0;
  l2 = alpha;
  double obj_l2_b = 0.5721784062720949;

  fx = run(handle, loss_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_b, fx));

  fx = run_api(cuml_handle, 2, C, loss_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_b, fx));

  l1 = alpha;
  l2 = 0.0;
  double obj_l1_no_b = 0.6606929813245878;

  fx = run(handle, loss_no_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_no_b, fx));

  fx = run_api(cuml_handle, 2, C, loss_no_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_no_b, fx));

  l1 = 0.0;
  l2 = alpha;

  double obj_l2_no_b = 0.6597171282106854;

  fx = run(handle, loss_no_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_no_b, fx));

  fx = run_api(cuml_handle, 2, C, loss_no_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_no_b, fx));
}

TEST_F(QuasiNewtonTest, linear_regression_vs_sklearn) {
  CompareApprox<double> compApprox(tol);
  double y[N] = {0.2675836026202781,  -0.0678277759663704, -0.6334027174275105,
                 -0.1018336189077367, 0.0933815935886932,  -1.1058853496996381,
                 -0.1658298189619160, -0.2954290675648911, 0.7966520536712608,
                 -1.0767450516284769};
  updateDeviceAsync(ydev->data, &y[0], ydev->len, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  double fx, l1, l2;
  double alpha = 0.01;

  SimpleVecOwning<double> w0(allocator, D + 1, stream);
  SimpleVecOwning<double> z(allocator, N, stream);
  SquaredLoss<double> loss_b(handle, D, true);
  SquaredLoss<double> loss_no_b(handle, D, false);

  l1 = alpha;
  l2 = 0.0;
  double w_l1_b[2] = {-0.4952397281519840, 0.3813315300180231};
  double b_l1_b = -0.08140861819001188;
  double obj_l1_b = 0.011136986298775138;
  fx = run(handle, loss_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l1_b[0], &b_l1_b, w0.data, loss_b,
                               compApprox, stream));

  fx = run_api(cuml_handle, 1, 1, loss_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_b, fx));

  l1 = 0.0;
  l2 = alpha;
  double w_l2_b[2] = {-0.5022384743587150, 0.3937352417485087};
  double b_l2_b = -0.08062397391797513;
  double obj_l2_b = 0.004268621967866347;

  fx = run(handle, loss_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l2_b[0], &b_l2_b, w0.data, loss_b,
                               compApprox, stream));

  fx = run_api(cuml_handle, 1, 1, loss_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_b, fx));

  l1 = alpha;
  l2 = 0.0;
  double w_l1_no_b[2] = {-0.5175178128147135, 0.3720844589831813};
  double obj_l1_no_b = 0.013981355746112447;

  fx = run(handle, loss_no_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_no_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l1_no_b[0], nobptr, w0.data,
                               loss_no_b, compApprox, stream));

  fx = run_api(cuml_handle, 1, 1, loss_no_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l1_no_b, fx));

  l1 = 0.0;
  l2 = alpha;
  double w_l2_no_b[2] = {-0.5241651041233270, 0.3846317886627560};
  double obj_l2_no_b = 0.007061261366969662;

  fx = run(handle, loss_no_b, *Xdev, *ydev, l1, l2, w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_no_b, fx));
  ASSERT_TRUE(checkParamsEqual(handle, &w_l2_no_b[0], nobptr, w0.data,
                               loss_no_b, compApprox, stream));

  fx = run_api(cuml_handle, 1, 1, loss_no_b.fit_intercept, *Xdev, *ydev, l1, l2,
               w0.data, z, 0, stream);
  ASSERT_TRUE(compApprox(obj_l2_no_b, fx));
}

TEST_F(QuasiNewtonTest, dense_vs_sparse) {
  hipStream_t stream = handle_ptr->getStream();
  const cumlHandle_impl &handle = handle_ptr->getImpl();

  CompareApprox<double> compApprox(tol);
  // Test case generated in python and solved with sklearn
  double yhost[10] = {1, 1, 1, 0, 1, 0, 1, 0, 1, 0};

  std::vector<double> Xsparsified(N * D, 0);
  int nnz = 0;
  const double *Xptr = &X[0][0];
  for (int it = 0; it < N * D; it++) {
    if (std::abs(Xptr[it]) < 0.5) {
      Xsparsified[it] = Xptr[it];
      nnz++;
    }
  }
  SimpleMatOwning<double> X(handle, N, D, stream, COL_MAJOR);
  updateDevice(X.data, &Xsparsified[0], X.len);
  SimpleVecOwning<double> y(handle, N, stream);
  updateDevice(y.data, &yhost[0], y.len);
  SimpleVecOwning<double> csrVal(handle, nnz, stream);
  SimpleVecOwning<int> csrRowPtr(handle, N + 1, stream);
  SimpleVecOwning<int> csrColInd(handle, nnz, stream);

  SimpleVecOwning<int> nnzPerRow(handle, N, stream);
  SimpleMatOwning<double> tmp(handle,1, N, stream);
  int nnzTotal;

  hipsparseMatDescr_t descr;
  CUSPARSE_CHECK(hipsparseCreateMatDescr(&descr));
  hipsparseDnnz(handle.getcusparseHandle(), HIPSPARSE_DIRECTION_ROW, N, D, descr,
               X.data, N, nnzPerRow.data, &nnzTotal);

  hipsparseDdense2csr(handle.getcusparseHandle(), N, D, descr, X.data, N,
                     nnzPerRow.data, csrVal.data, csrRowPtr.data,
                     csrColInd.data);

  LogisticLoss<double> logLoss(handle, D, false);
  GLMWithData<double, SimpleMat<double>, decltype(logLoss)> lossDense(
      &logLoss, X, y, tmp);
  LBFGSParam<double> opt_param;
  opt_param.epsilon = 1e-5;
  opt_param.max_iterations = 100;
  opt_param.m = 2;
  opt_param.max_linesearch = 50;
  SimpleVecOwning<double> w(handle, logLoss.n_param, stream);
  int verbosity = 1;

  double fxd, fxs;
  int num_iters;
  double l1 = 0.001;
  qn_minimize(handle, w, &fxd, &num_iters, lossDense, l1, opt_param, stream,
              verbosity);

  CsrMat<double> csr(csrVal.data, csrRowPtr.data, csrColInd.data, N, D, nnz);
  GLMWithData<double, CsrMat<double>, decltype(logLoss)> lossSparse(
      &logLoss, csr, y, tmp);

  w.fill(0, stream);
  qn_minimize(handle, w, &fxs, &num_iters, lossSparse, l1, opt_param, stream,
              verbosity);

  ASSERT_TRUE(compApprox(fxd, fxs));
}

} // namespace GLM
} // end namespace ML
