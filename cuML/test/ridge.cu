#include "glm/ridge.h"
#include <gtest/gtest.h>
#include <cuda_utils.h>
#include <test_utils.h>
#include "ml_utils.h"

namespace ML {
namespace GLM {

using namespace MLCommon;

template<typename T>
struct RidgeInputs {
	T tol;
	int n_row;
	int n_col;
	int n_row_2;
	int algo;
	T alpha;
};

template<typename T>
class RidgeTest: public ::testing::TestWithParam<RidgeInputs<T> > {
protected:
	void basicTest() {
		params = ::testing::TestWithParam<RidgeInputs<T>>::GetParam();
		int len = params.n_row * params.n_col;
		int len2 = params.n_row_2 * params.n_col;

		hipblasHandle_t cublas_handle;
		CUBLAS_CHECK(hipblasCreate(&cublas_handle));

		hipsolverHandle_t cusolver_handle = NULL;
		CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));

		allocate(data, len);
		allocate(labels, params.n_row);
		allocate(coef, params.n_col);
		allocate(coef2, params.n_col);
		allocate(coef3, params.n_col);
		allocate(coef_ref, params.n_col);
		allocate(coef2_ref, params.n_col);
		allocate(coef3_ref, params.n_col);
		allocate(pred_data, len2);
		allocate(pred, params.n_row_2);
		allocate(pred_ref, params.n_row_2);
		allocate(pred2, params.n_row_2);
		allocate(pred2_ref, params.n_row_2);
		allocate(pred3, params.n_row_2);
		allocate(pred3_ref, params.n_row_2);
		T alpha = params.alpha;

		T data_h[len] = { 0.0, 0.0, 1.0, 0.0, 0.0, 1.0 };
		updateDevice(data, data_h, len);

		T labels_h[params.n_row] = { 0.0, 0.1, 1.0 };
		updateDevice(labels, labels_h, params.n_row);

		T coef_ref_h[params.n_col] = { 0.39999998, 0.4 };
		updateDevice(coef_ref, coef_ref_h, params.n_col);

		T coef2_ref_h[params.n_col] = { 0.3454546 , 0.34545454 };
		updateDevice(coef2_ref, coef2_ref_h, params.n_col);

		T coef3_ref_h[params.n_col] = { 0.3799999 , 0.38000008 };
		updateDevice(coef3_ref, coef3_ref_h, params.n_col);

		T pred_data_h[len2] = { 0.5, 2.0, 0.2, 1.0 };
		updateDevice(pred_data, pred_data_h, len2);

		T pred_ref_h[params.n_row_2] = { 0.28, 1.1999999 };
		updateDevice(pred_ref, pred_ref_h, params.n_row_2);

		T pred2_ref_h[params.n_row_2] = { 0.37818184, 1.1727273 };
		updateDevice(pred2_ref, pred2_ref_h, params.n_row_2);

		T pred3_ref_h[params.n_row_2] = { 0.37933332, 1.2533332 };
		updateDevice(pred3_ref, pred3_ref_h, params.n_row_2);

		intercept = T(0);

		ridgeFit(data, params.n_row, params.n_col, labels, &alpha, 1, coef,
				&intercept, false, false, cublas_handle,
				cusolver_handle, params.algo);

		ridgePredict(pred_data, params.n_row_2, params.n_col, coef, intercept, pred,
				cublas_handle);


		updateDevice(data, data_h, len);
		updateDevice(labels, labels_h, params.n_row);

		intercept2 = T(0);
		ridgeFit(data, params.n_row, params.n_col, labels, &alpha, 1, coef2,
				&intercept2, true, false, cublas_handle,
						cusolver_handle, params.algo);

		ridgePredict(pred_data, params.n_row_2, params.n_col, coef2, intercept2, pred2,
						cublas_handle);


		updateDevice(data, data_h, len);
		updateDevice(labels, labels_h, params.n_row);

		intercept3 = T(0);
		ridgeFit(data, params.n_row, params.n_col, labels, &alpha, 1, coef3,
				&intercept3, true, true, cublas_handle,
				cusolver_handle, params.algo);

		ridgePredict(pred_data, params.n_row_2, params.n_col, coef3, intercept3, pred3,
				   cublas_handle);



		CUBLAS_CHECK(hipblasDestroy(cublas_handle));
		CUSOLVER_CHECK(hipsolverDnDestroy(cusolver_handle));

	}

	void SetUp() override {
		basicTest();
	}

	void TearDown() override {
		CUDA_CHECK(hipFree(data));
		CUDA_CHECK(hipFree(labels));
		CUDA_CHECK(hipFree(coef));
		CUDA_CHECK(hipFree(coef_ref));
		CUDA_CHECK(hipFree(coef2));
		CUDA_CHECK(hipFree(coef2_ref));
		CUDA_CHECK(hipFree(coef3));
		CUDA_CHECK(hipFree(coef3_ref));
		CUDA_CHECK(hipFree(pred_data));
		CUDA_CHECK(hipFree(pred));
		CUDA_CHECK(hipFree(pred_ref));
		CUDA_CHECK(hipFree(pred2));
		CUDA_CHECK(hipFree(pred2_ref));
		CUDA_CHECK(hipFree(pred3));
		CUDA_CHECK(hipFree(pred3_ref));

	}

protected:
	RidgeInputs<T> params;
	T *data, *labels, *coef, *coef_ref, *pred_data, *pred, *pred_ref;
	T *coef2, *coef2_ref, *pred2, *pred2_ref;
	T *coef3, *coef3_ref, *pred3, *pred3_ref;
	T intercept, intercept2, intercept3;
};

const std::vector<RidgeInputs<float> > inputsf2 = {
		{ 0.001f, 3, 2, 2, 0, 0.5f },
		{ 0.001f, 3, 2, 2, 1, 0.5f } };

const std::vector<RidgeInputs<double> > inputsd2 = {
		{ 0.001, 3, 2, 2, 0, 0.5 },
		{ 0.001, 3, 2, 2, 1, 0.5 } };

typedef RidgeTest<float> RidgeTestF;
TEST_P(RidgeTestF, Fit) {

	ASSERT_TRUE(
			devArrMatch(coef_ref, coef, params.n_col,
					CompareApproxAbs<float>(params.tol)));

	ASSERT_TRUE(
			devArrMatch(coef2_ref, coef2, params.n_col,
				    CompareApproxAbs<float>(params.tol)));


	ASSERT_TRUE(
			devArrMatch(coef3_ref, coef3, params.n_col,
					CompareApproxAbs<float>(params.tol)));

	ASSERT_TRUE(
			devArrMatch(pred_ref, pred, params.n_row_2,
					CompareApproxAbs<float>(params.tol)));

	ASSERT_TRUE(
			devArrMatch(pred2_ref, pred2, params.n_row_2,
					CompareApproxAbs<float>(params.tol)));


	ASSERT_TRUE(
			devArrMatch(pred3_ref, pred3, params.n_row_2,
					CompareApproxAbs<float>(params.tol)));
}

typedef RidgeTest<double> RidgeTestD;
TEST_P(RidgeTestD, Fit) {

	ASSERT_TRUE(
			devArrMatch(coef_ref, coef, params.n_col,
					CompareApproxAbs<double>(params.tol)));

	ASSERT_TRUE(
				devArrMatch(coef2_ref, coef2, params.n_col,
					CompareApproxAbs<double>(params.tol)));


	ASSERT_TRUE(
				devArrMatch(coef3_ref, coef3, params.n_col,
					CompareApproxAbs<double>(params.tol)));

	ASSERT_TRUE(
			devArrMatch(pred_ref, pred, params.n_row_2,
					CompareApproxAbs<double>(params.tol)));

	ASSERT_TRUE(
			devArrMatch(pred2_ref, pred2, params.n_row_2,
					CompareApproxAbs<double>(params.tol)));


	ASSERT_TRUE(
			devArrMatch(pred3_ref, pred3, params.n_row_2,
					CompareApproxAbs<double>(params.tol)));
}

INSTANTIATE_TEST_CASE_P(RidgeTests, RidgeTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(RidgeTests, RidgeTestD, ::testing::ValuesIn(inputsd2));

}
} // end namespace ML
