/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kmeans/kmeans.cu"
#include <vector>
#include <gtest/gtest.h>
#include <cuda_utils.h>
#include <test_utils.h>

namespace ML {

using namespace MLCommon;

template<typename T>
struct KmeansInputs {
    int n_clusters;
	T tol;
	int n_row;
	int n_col;
};

template<typename T>
class KmeansTest: public ::testing::TestWithParam<KmeansInputs<T> > {
protected:
	void basicTest() {
		params = ::testing::TestWithParam<KmeansInputs<T>>::GetParam();
        int m = params.n_row;
        int n = params.n_col;
        int k = params.n_clusters;

        // make space for outputs : pred_centroids, pred_labels
        // and reference output : labels_ref
        allocate(d_srcdata, n * m);
   		allocate(labels_fit, m);
   		allocate(labels_ref_fit, m);
        allocate(pred_centroids, k * n);
        allocate(centroids_ref, k * n);

        // make testdata on host
        std::vector<T> h_srcdata = {1.0,1.0,3.0,4.0, 1.0,2.0,2.0,3.0};
        h_srcdata.resize(n * m);
        updateDevice(d_srcdata, h_srcdata.data(), m*n);

        // make and assign reference output
        std::vector<int> h_labels_ref_fit = {1, 1, 0, 0};
        h_labels_ref_fit.resize(m);
        updateDevice(labels_ref_fit, h_labels_ref_fit.data(), m);

        std::vector<T> h_centroids_ref = {3.5,2.5, 1.0,1.5};
        h_centroids_ref.resize(k * n);
        updateDevice(centroids_ref, h_centroids_ref.data(), k * n);

        // The actual kmeans api calls
        // fit
        make_ptr_kmeans(0, verbose, seed, gpu_id, n_gpu, m, n,
            ord, k, k, max_iterations,
            init_from_data, params.tol, d_srcdata, nullptr, pred_centroids, labels_fit);
    }

 	void SetUp() override {
		basicTest();
	}

	void TearDown() override {
        CUDA_CHECK(hipFree(d_srcdata));
		CUDA_CHECK(hipFree(labels_fit));
		CUDA_CHECK(hipFree(pred_centroids));
		CUDA_CHECK(hipFree(labels_ref_fit));
		CUDA_CHECK(hipFree(centroids_ref));

	}

protected:
	KmeansInputs<T> params;
	T *d_srcdata;
	int *labels_fit, *labels_ref_fit;
    T *pred_centroids, *centroids_ref;
    int verbose = 0;
    int seed = 1;
    int gpu_id = 0;
    int n_gpu = -1;
    char ord = 'c'; // here c means col order, NOT C (vs F) order
    int max_iterations = 300;
    int init_from_data = 0;
};

const std::vector<KmeansInputs<float> > inputsf2 = {
		{ 2, 0.05f, 4, 2 }};

const std::vector<KmeansInputs<double> > inputsd2 = {
		{ 2, 0.05, 4, 2 }};


// FIXME: These tests are disabled due to being too sensitive to RNG:
// https://github.com/rapidsai/cuml/issues/71
typedef KmeansTest<float> KmeansTestF;
TEST_P(KmeansTestF, DISABLED_Fit) {
	ASSERT_TRUE(
			devArrMatch(labels_ref_fit, labels_fit, params.n_row,
					CompareApproxAbs<float>(params.tol)));
	ASSERT_TRUE(
			devArrMatch(centroids_ref, pred_centroids, params.n_clusters * params.n_col,
					CompareApproxAbs<float>(params.tol)));
}

typedef KmeansTest<double> KmeansTestD;
TEST_P(KmeansTestD, DISABLED_Fit) {
	ASSERT_TRUE(
			devArrMatch(labels_ref_fit, labels_fit, params.n_row,
					CompareApproxAbs<double>(params.tol)));
	ASSERT_TRUE(
			devArrMatch(centroids_ref, pred_centroids, params.n_clusters * params.n_col,
					CompareApproxAbs<double>(params.tol)));
}

INSTANTIATE_TEST_CASE_P(KmeansTests, KmeansTestF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_CASE_P(KmeansTests, KmeansTestD, ::testing::ValuesIn(inputsd2));

} // end namespace ML
