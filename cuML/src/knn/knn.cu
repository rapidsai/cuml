/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_utils.h"
#include "knn.h"
#include <hip/hip_runtime.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/Heap.h>

#include <vector>
#include <sstream>


namespace ML {


	/**
	 * Build a kNN object for training and querying a k-nearest neighbors model.
	 * @param D 	number of features in each vector
	 */
	kNN::kNN(int D): D(D), total_n(0), indices(0){}
	kNN::~kNN() {

		for(faiss::gpu::GpuIndexFlatL2* idx : sub_indices) {
			delete idx;
		}

		for(faiss::gpu::GpuResources *r : res) {
			delete r;
		}
	}

	/**
	 * Fit a kNN model by creating separate indices for multiple given
	 * instances of kNNParams.
	 * @param input  an array of pointers to data on (possibly different) devices
	 * @param N 	 number of items in input array.
	 */
	void kNN::fit(kNNParams *input, int N) {

		for(int i = 0; i < N; i++) {

			kNNParams *params = &input[i];

			hipPointerAttribute_t att;
			hipError_t err = hipPointerGetAttributes(&att, params->ptr);

			if(err == 0 && att.device > -1) {

				if(i < N)
					id_ranges.push_back(total_n);

				this->total_n += params->N;
				this->indices += 1;

				res.emplace_back(new faiss::gpu::StandardGpuResources());

				faiss::gpu::GpuIndexFlatConfig config;
				config.device = att.device;
				config.useFloat16 = false;
				config.storeTransposed = false;

				sub_indices.emplace_back(
						new faiss::gpu::GpuIndexFlatL2(res[i], D, config)
				);

				// It's only necessary to maintain our set of shards because
				// the GpuIndexFlat class does not support add_with_ids(),
				// a dependency of the IndexShards composite class.
				// As a result, we need to add the ids ourselves
				// and have the reducer/combiner re-label the indices
				// based on the shards they came from.
				sub_indices[i]->add(params->N, params->ptr);
			} else {
				std::stringstream ss;
				ss << "Input memory for " << &params << " failed. isDevice?=" << att.devicePointer;
				throw ss.str();
			}
		}
	}

	/**
	 * Search the kNN for the k-nearest neighbors of a set of query vectors
	 * @param search_items set of vectors to query for neighbors
	 * @param n 		   number of items in search_items
	 * @param res_I 	   pointer to device memory for returning k nearest indices
	 * @param res_D		   pointer to device memory for returning k nearest distances
	 * @param k			   number of neighbors to query
	 */
	void kNN::search(const float *search_items, int n,
			long *res_I, float *res_D, int k) {

		float *result_D = new float[k*n];
		long *result_I = new long[k*n];

		float *all_D = new float[indices*k*n];
		long *all_I = new long[indices*k*n];

        for(int i = 0; i < indices; i++)
			this->sub_indices[i]->search(n, search_items, k,
					all_D+(i*k*n), all_I+(i*k*n));

		merge_tables<faiss::CMin<float, int>>(n, k, indices,
				result_D, result_I, all_D, all_I, id_ranges.data());

		MLCommon::updateDevice(res_D, result_D, k*n, 0);
		MLCommon::updateDevice(res_I, result_I, k*n, 0);

		delete all_D;
		delete all_I;

		delete result_D;
		delete result_I;
	}


	/** Merge results from several shards into a single result set.
	 * @param all_distances  size nshard * n * k
	 * @param all_labels     idem
	 * @param translartions  label translations to apply, size nshard
	 */
	template <class C>
	void kNN::merge_tables (long n, long k, long nshard,
					   float *distances, long *labels,
					   float *all_distances,
					   long *all_labels,
					   long *translations) {
		if(k == 0) {
			return;
		}

		long stride = n * k;
		#pragma omp parallel
		{
			std::vector<int> buf (2 * nshard);
			int * pointer = buf.data();
			int * shard_ids = pointer + nshard;
			std::vector<float> buf2 (nshard);
			float * heap_vals = buf2.data();
			#pragma omp for
			for (long i = 0; i < n; i++) {
				// the heap maps values to the shard where they are
				// produced.
				const float *D_in = all_distances + i * k;
				const long *I_in = all_labels + i * k;
				int heap_size = 0;

				for (long s = 0; s < nshard; s++) {
					pointer[s] = 0;
					if (I_in[stride * s] >= 0)
						heap_push<C> (++heap_size, heap_vals, shard_ids,
									 D_in[stride * s], s);
				}

				float *D = distances + i * k;
				long *I = labels + i * k;

				for (int j = 0; j < k; j++) {
					if (heap_size == 0) {
						I[j] = -1;
						D[j] = C::neutral();
					} else {
						// pop best element
						int s = shard_ids[0];
						int & p = pointer[s];
						D[j] = heap_vals[0];
						I[j] = I_in[stride * s + p] + translations[s];

						heap_pop<C> (heap_size--, heap_vals, shard_ids);
						p++;
						if (p < k && I_in[stride * s + p] >= 0)
							heap_push<C> (++heap_size, heap_vals, shard_ids,
										 D_in[stride * s + p], s);
					}
				}
			}
		}

	};

};


// end namespace ML
