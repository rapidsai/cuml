#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "utils.h"
#include <cuML_api.h>
#include "runner.h"
#include "dbscan.hpp"
#include "dbscan.h"
#include <common/cumlHandle.hpp>

namespace ML {

using namespace Dbscan;


void dbscanFit(const cumlHandle& handle, float *input, int n_rows, int n_cols, float eps, int min_pts,
		       int *labels) {
	dbscanFitImpl(handle.getImpl(), input, n_rows, n_cols, eps, min_pts, labels, handle.getStream());
}

void dbscanFit(const cumlHandle& handle, double *input, int n_rows, int n_cols, double eps, int min_pts,
		       int *labels) {
	dbscanFitImpl(handle.getImpl(), input, n_rows, n_cols, eps, min_pts, labels, handle.getStream());
}

// Following are two versions of dbscanFit, that do not take cumlHandle as
// input arguments. The cumlHandle is created inside dbscanFit on each new call.

void dbscanFit(float *input, int n_rows, int n_cols, float eps, int min_pts,
               int *labels) {
    cumlHandle handle;
    dbscanFitImpl(handle.getImpl(), input, n_rows, n_cols, eps, min_pts, labels, handle.getStream());
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void dbscanFit(double *input, int n_rows, int n_cols, double eps, int min_pts,
               int *labels) {

    cumlHandle handle;
    dbscanFitImpl(handle.getImpl(), input, n_rows, n_cols, eps, min_pts, labels, handle.getStream());
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}
/** @} */

};
// end namespace ML
extern "C" cumlError_t cumlSpDbscanFit(cumlHandle_t handle, float *input, int n_rows, int n_cols, float eps, int min_pts,
               int *labels) {
    cumlError_t status;
    ML::cumlHandle *handle_ptr;
    std::tie(handle_ptr, status) = ML::handleMap.lookupHandlePointer(handle);
    if (status == CUML_SUCCESS) {
        try
        {
            dbscanFit(*handle_ptr, input, n_rows, n_cols, eps, min_pts, labels);
        }
        //TODO: Implement this
        //catch (const MLCommon::Exception& e)
        //{
        //    //log e.what()?
        //    status =  e.getErrorCode();
        //}
        catch (...)
        {
            status = CUML_ERROR_UNKNOWN;
        }
    }
    return status;

}

extern "C" cumlError_t cumlDpDbscanFit(cumlHandle_t handle, double *input, int n_rows, int n_cols, double eps, int min_pts,
               int *labels) {
    cumlError_t status;
    ML::cumlHandle *handle_ptr;
    std::tie(handle_ptr, status) = ML::handleMap.lookupHandlePointer(handle);
    if (status == CUML_SUCCESS) {
        try
        {
            dbscanFit(*handle_ptr, input, n_rows, n_cols, eps, min_pts, labels);
        }
        //TODO: Implement this
        //catch (const MLCommon::Exception& e)
        //{
        //    //log e.what()?
        //    status =  e.getErrorCode();
        //}
        catch (...)
        {
            status = CUML_ERROR_UNKNOWN;
        }
    }
    return status;
}
