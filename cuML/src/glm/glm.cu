#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "ols.h"
#include "ridge.h"
#include "glm.hpp"
#include "glm/qn/qn.h"
#include "cuML.hpp"
#include "glm/glm_api.h"

namespace ML {
namespace GLM {

using namespace MLCommon;

void olsFit(float *input, int n_rows, int n_cols, float *labels, float *coef,
            float *intercept, bool fit_intercept, bool normalize, int algo) {
    cumlHandle handle;
    olsFit(handle.getImpl(), input, n_rows, n_cols, labels, coef, intercept, fit_intercept,
           normalize, handle.getStream(), algo);
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void olsFit(double *input, int n_rows, int n_cols, double *labels, double *coef,
		double *intercept, bool fit_intercept, bool normalize, int algo) {
    cumlHandle handle;
    olsFit(handle.getImpl(), input, n_rows, n_cols, labels, coef, intercept, fit_intercept,
           normalize, handle.getStream(), algo);
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void olsPredict(const float *input, int n_rows, int n_cols, const float *coef,
		float intercept, float *preds) {
    cumlHandle handle;
    olsPredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds,
               handle.getStream());
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void olsPredict(const double *input, int n_rows, int n_cols, const double *coef,
		double intercept, double *preds) {
    cumlHandle handle;
    olsPredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds,
               handle.getStream());
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgeFit(float *input, int n_rows, int n_cols, float *labels, float *alpha,
		int n_alpha, float *coef, float *intercept, bool fit_intercept,
		bool normalize, int algo) {
    cumlHandle handle;
    ridgeFit(handle.getImpl(), input, n_rows, n_cols, labels, alpha, n_alpha, coef, intercept,
             fit_intercept, normalize, handle.getStream(), algo);
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgeFit(double *input, int n_rows, int n_cols, double *labels,
		double *alpha, int n_alpha, double *coef, double *intercept,
		bool fit_intercept, bool normalize, int algo) {
    cumlHandle handle;
    ridgeFit(handle.getImpl(), input, n_rows, n_cols, labels, alpha, n_alpha, coef, intercept,
             fit_intercept, normalize, handle.getStream(), algo);
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgePredict(const float *input, int n_rows, int n_cols, const float *coef,
		float intercept, float *preds) {
    cumlHandle handle;
    ridgePredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds, handle.getStream());
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void ridgePredict(const double *input, int n_rows, int n_cols, const double *coef,
		double intercept, double *preds) {
    cumlHandle handle;
    ridgePredict(handle.getImpl(), input, n_rows, n_cols, coef, intercept, preds, handle.getStream());
    ///@todo this should go away after cumlHandle exposure in the interface
    CUDA_CHECK(hipStreamSynchronize(handle.getStream()));
}

void qnFit(const cumlHandle &cuml_handle, float *X, float *y, int N, int D,
           int C, bool fit_intercept, float l1, float l2, int max_iter,
           float grad_tol, int linesearch_max_iter, int lbfgs_memory,
           int verbosity, float *w0, float *f, int *num_iters, bool X_col_major,
           int loss_type) {

  qnFit(cuml_handle.getImpl(), X, y, N, D, C, fit_intercept, l1, l2, max_iter,
        grad_tol, linesearch_max_iter, lbfgs_memory, verbosity, w0, f,
        num_iters, X_col_major, loss_type, cuml_handle.getStream());
}

void qnFit(const cumlHandle &cuml_handle, double *X, double *y, int N, int D,
           int C, bool fit_intercept, double l1, double l2, int max_iter,
           double grad_tol, int linesearch_max_iter, int lbfgs_memory,
           int verbosity, double *w0, double *f, int *num_iters,
           bool X_col_major, int loss_type) {

  qnFit(cuml_handle.getImpl(), X, y, N, D, C, fit_intercept, l1, l2, max_iter,
        grad_tol, linesearch_max_iter, lbfgs_memory, verbosity, w0, f,
        num_iters, X_col_major, loss_type, cuml_handle.getStream());
}

void qnPredict(const cumlHandle &cuml_handle, float *X, int N, int D, int C,
               bool fit_intercept, float *params, bool X_col_major,
               int loss_type, float *preds) {
  qnPredict(cuml_handle.getImpl(), X, N, D, C, fit_intercept, params,
            X_col_major, loss_type, preds, cuml_handle.getStream());
}

void qnPredict(const cumlHandle &cuml_handle, double *X, int N, int D, int C,
               bool fit_intercept, double *params, bool X_col_major,
               int loss_type, double *preds) {
  qnPredict(cuml_handle.getImpl(), X, N, D, C, fit_intercept, params,
            X_col_major, loss_type, preds, cuml_handle.getStream());
}

} // namespace GLM
} // namespace ML

extern "C" cumlError_t cumlSpQnFit(cumlHandle_t cuml_handle, float *X, float *y,
                                   int N, int D, int C, bool fit_intercept,
                                   float l1, float l2, int max_iter,
                                   float grad_tol, int linesearch_max_iter,
                                   int lbfgs_memory, int verbosity, float *w0,
                                   float *f, int *num_iters, bool X_col_major,
                                   int loss_type) {
  cumlError_t status;
  ML::cumlHandle *handle_ptr;
  std::tie(handle_ptr, status) = ML::handleMap.lookupHandlePointer(cuml_handle);
  if (status == CUML_SUCCESS) {
    try {
      ML::GLM::qnFit(*handle_ptr, X, y, N, D, C, fit_intercept, l1, l2,
                     max_iter, grad_tol, linesearch_max_iter, lbfgs_memory,
                     verbosity, w0, f, num_iters, X_col_major, loss_type);

    }
    // TODO: Implement this
    // catch (const MLCommon::Exception& e)
    //{
    //    //log e.what()?
    //    status =  e.getErrorCode();
    //}
    catch (...) {
      status = CUML_ERROR_UNKNOWN;
    }
  }
  return status;
}

extern "C" cumlError_t
cumlDpQnFit(cumlHandle_t cuml_handle, double *X, double *y, int N, int D, int C,
            bool fit_intercept, double l1, double l2, int max_iter,
            double grad_tol, int linesearch_max_iter, int lbfgs_memory,
            int verbosity, double *w0, double *f, int *num_iters,
            bool X_col_major, int loss_type) {
  cumlError_t status;
  ML::cumlHandle *handle_ptr;
  std::tie(handle_ptr, status) = ML::handleMap.lookupHandlePointer(cuml_handle);
  if (status == CUML_SUCCESS) {
    try {
      ML::GLM::qnFit(*handle_ptr, X, y, N, D, C, fit_intercept, l1, l2,
                     max_iter, grad_tol, linesearch_max_iter, lbfgs_memory,
                     verbosity, w0, f, num_iters, X_col_major, loss_type);

    }
    // TODO: Implement this
    // catch (const MLCommon::Exception& e)
    //{
    //    //log e.what()?
    //    status =  e.getErrorCode();
    //}
    catch (...) {
      status = CUML_ERROR_UNKNOWN;
    }
  }
  return status;
}
