/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "ols.h"
#include "ridge.h"
#include "glm_c.h"

namespace ML {
namespace GLM {

using namespace MLCommon;

void olsFit(float *input, int n_rows, int n_cols, float *labels, float *coef,
		float *intercept, bool fit_intercept, bool normalize, int algo) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	hipsolverHandle_t cusolver_handle = NULL;
	CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));

	olsFit(input, n_rows, n_cols, labels, coef, intercept, fit_intercept,
			normalize, cublas_handle, cusolver_handle, algo);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));
	CUSOLVER_CHECK(hipsolverDnDestroy(cusolver_handle));

}

void olsFit(double *input, int n_rows, int n_cols, double *labels, double *coef,
		double *intercept, bool fit_intercept, bool normalize, int algo) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	hipsolverHandle_t cusolver_handle = NULL;
	CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));

	olsFit(input, n_rows, n_cols, labels, coef, intercept, fit_intercept,
			normalize, cublas_handle, cusolver_handle, algo);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));
	CUSOLVER_CHECK(hipsolverDnDestroy(cusolver_handle));

}

void olsPredict(const float *input, int n_rows, int n_cols, const float *coef,
		float intercept, float *preds) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	olsPredict(input, n_rows, n_cols, coef, intercept, preds, cublas_handle);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));

}

void olsPredict(const double *input, int n_rows, int n_cols, const double *coef,
		double intercept, double *preds) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	olsPredict(input, n_rows, n_cols, coef, intercept, preds, cublas_handle);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));

}

void ridgeFit(float *input, int n_rows, int n_cols, float *labels, float *alpha,
		int n_alpha, float *coef, float *intercept, bool fit_intercept,
		bool normalize, int algo) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	hipsolverHandle_t cusolver_handle = NULL;
	CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));

	ridgeFit(input, n_rows, n_cols, labels, alpha, n_alpha, coef, intercept,
			fit_intercept, normalize, cublas_handle, cusolver_handle, algo);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));
	CUSOLVER_CHECK(hipsolverDnDestroy(cusolver_handle));

}

void ridgeFit(double *input, int n_rows, int n_cols, double *labels,
		double *alpha, int n_alpha, double *coef, double *intercept,
		bool fit_intercept, bool normalize, int algo) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	hipsolverHandle_t cusolver_handle = NULL;
	CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));

	ridgeFit(input, n_rows, n_cols, labels, alpha, n_alpha, coef, intercept,
			fit_intercept, normalize, cublas_handle, cusolver_handle, algo);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));
	CUSOLVER_CHECK(hipsolverDnDestroy(cusolver_handle));

}

void ridgePredict(const float *input, int n_rows, int n_cols, const float *coef,
		float intercept, float *preds) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	ridgePredict(input, n_rows, n_cols, coef, intercept, preds, cublas_handle);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));

}

void ridgePredict(const double *input, int n_rows, int n_cols, const double *coef,
		double intercept, double *preds) {

	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));

	ridgePredict(input, n_rows, n_cols, coef, intercept, preds, cublas_handle);

	CUBLAS_CHECK(hipblasDestroy(cublas_handle));

}

}
}
